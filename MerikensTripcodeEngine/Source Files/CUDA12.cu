#include "hip/hip_runtime.h"
// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



///////////////////////////////////////////////////////////////////////////////
// INCLUDE FILE(S)                                                           //
///////////////////////////////////////////////////////////////////////////////

#include "MerikensTripcodeEngine.h"



#define CUDA_SHA1_MAX_PASS_COUNT            2048  // Be VERY CAREFUL when you change this constant.
#define CUDA_SHA1_NUM_THREADS_PER_BLOCK     128



///////////////////////////////////////////////////////////////////////////////
// VARIABLES FOR CUDA CODES                                                  //
///////////////////////////////////////////////////////////////////////////////

__device__ __constant__ unsigned char   cudaKeyCharTable_OneByte[SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   cudaKeyCharTable_FirstByte  [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   cudaKeyCharTable_SecondByte [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   cudaKeyCharTable_SecondByteAndOneByte[SIZE_KEY_CHAR_TABLE];
__device__ __constant__ char            CUDA_base64CharTable[64];
__device__ __constant__ unsigned char   CUDA_smallChunkBitmap[SMALL_CHUNK_BITMAP_SIZE];



///////////////////////////////////////////////////////////////////////////////
// SHA-1 HASH GENERATION ON CUDA DEVICE                                      //
///////////////////////////////////////////////////////////////////////////////

// Macros are used extensively for the sake of optimization.
// Good luck deciphering!

// The following are marcos and constants for SHA-1 hash generation.

// Circular left rotation of 32-bit value 'val' left by 'bits' bits
// (assumes that 'bits' is always within range from 0 to 32)
#define ROTL(bits, val) (((val) << (bits)) | ((val) >> (32 - (bits))))

// Central routine for calculating the hash value. See the FIPS
// 180-3 standard p. 17f for a detailed explanation.
#define f1 	( ( B & C ) ^ ( ( ~ B ) & D ) )
#define f2  ( B ^ C ^ D )
#define f3  ( ( B & C ) ^ ( B & D ) ^ ( C & D ) )
#define f4  f2

// Initial hash values (see p. 14 of FIPS 180-3)
#define H0 0x67452301
#define H1 0xefcdab89
#define H2 0x98badcfe
#define H3 0x10325476
#define H4 0xc3d2e1f0

// Constants required for hash calculation (see p. 11 of FIPS 180-3)
#define K0 0x5a827999
#define K1 0x6ed9eba1
#define K2 0x8f1bbcdc
#define K3 0xca62c1d6

#define SET_KEY_CHAR(var, flag, table, value)             \
	if (!(flag)) {                                        \
		var = (table)[(value)];                           \
		isSecondByte = IS_FIRST_BYTE_SJIS(var);           \
	} else {                                              \
		var = cudaKeyCharTable_SecondByte[(value)];          \
		isSecondByte = FALSE;                             \
	}                                                     \

#define ROUND_00_TO_19(t, w)                              \
		{                                                 \
			tmp = (ROTL(5, A) + f1 + E + (w) + K0);       \
			E = D;                                        \
			D = C;                                        \
			C = ROTL( 30, B );                            \
			B = A;                                        \
			A = tmp;                                      \
		}                                                 \

#define ROUND_20_TO_39(t, w)                              \
		{                                                 \
			tmp = (ROTL(5, A) + f2 + E + (w) + K1);       \
			E = D;                                        \
			D = C;                                        \
			C = ROTL( 30, B );                            \
			B = A;                                        \
			A = tmp;                                      \
		}                                                 \

#define ROUND_40_TO_59(t, w)                              \
		{                                                 \
			tmp = (ROTL(5, A) + f3 + E + (w) + K2);       \
			E = D;                                        \
			D = C;                                        \
			C = ROTL( 30, B );                            \
			B = A;                                        \
			A = tmp;                                      \
		}                                                 \

#define	ROUND_60_TO_79(t, w)                              \
		{                                                 \
			tmp = (ROTL(5, A) + f4 + E + (w) + K3 );      \
			E = D;                                        \
			D = C;                                        \
			C = ROTL( 30, B );                            \
			B = A;                                        \
			A = tmp;                                      \
		}                                                 \

#define CUDA_SHA1_DEFINE_SEARCH_FUNCTION(functionName) \
__global__ void (functionName)(\
	GPUOutput     *outputArray,\
	unsigned char *chunkBitmap,\
	unsigned int  *tripcodeChunkArray,\
	unsigned int   numTripcodeChunk,\
	unsigned char *keyAndRandomBytes\
) {

#define CUDA_SHA1_BEFORE_SEARCHING                                                                         \
	unsigned int        A, B, C, D, E, tmp;                                                                \
	unsigned char       key0, key1, key2, key3, key11;                                                     \
	unsigned char       found = 0;                                                                         \
	BOOL                isSecondByte = FALSE;                                                              \
	unsigned char      *tableForKey2;                                                                      \
	GPUOutput          *output = &outputArray[blockIdx.x * CUDA_SHA1_NUM_THREADS_PER_BLOCK + threadIdx.x]; \
    int                 passCount;                                                                         \
	int                 randomByte2 = keyAndRandomBytes[2];                                                         \
	int                 randomByte3 = keyAndRandomBytes[3];                                                         \
	                                                                                                       \
	output->numMatchingTripcodes = 0;                                                                      \
	SET_KEY_CHAR(key0, isSecondByte, cudaKeyCharTable_FirstByte, keyAndRandomBytes[0] + (blockIdx.x >> 6));        \
	SET_KEY_CHAR(key1, isSecondByte, cudaKeyCharTable_FirstByte, keyAndRandomBytes[1] + (threadIdx.x & 0x3f));     \
	tableForKey2 = (isSecondByte) ? (cudaKeyCharTable_SecondByte) : (cudaKeyCharTable_FirstByte);        \
	key11 = cudaKeyCharTable_SecondByteAndOneByte[keyAndRandomBytes[11] + (blockIdx.x & 0x3f)];                    \
	                                                                                                       \
	__shared__ unsigned int PW[80+1];                                                                        \
	__shared__ unsigned char smallChunkBitmap[SMALL_CHUNK_BITMAP_SIZE];                                        \
	if (threadIdx.x == 0) {                                                                                \
		PW[0]  = 0;                                                                                        \
		PW[1]  = (keyAndRandomBytes[4] << 24) | (keyAndRandomBytes[5] << 16) | (keyAndRandomBytes[ 6] << 8) | keyAndRandomBytes[ 7];           \
		PW[2]  = (keyAndRandomBytes[8] << 24) | (keyAndRandomBytes[9] << 16) | (keyAndRandomBytes[10] << 8) | key11;                  \
		PW[3]  = 0x80000000;                                                                               \
		PW[4]  = 0;                                                                                        \
		PW[5]  = 0;                                                                                        \
		PW[6]  = 0;                                                                                        \
		PW[7]  = 0;                                                                                        \
		PW[8]  = 0;                                                                                        \
		PW[9]  = 0;                                                                                        \
		PW[10] = 0;                                                                                        \
		PW[11] = 0;                                                                                        \
		PW[12] = 0;                                                                                        \
		PW[13] = 0;                                                                                        \
		PW[14] = 0;                                                                                        \
		PW[15] = 12 * 8;                                                                                   \
		PW[16] = ROTL(1, PW[16 - 3] ^ PW[16 - 8] ^ PW[16 - 14]);                                           \
		for (int t = 17; t < 80; ++t)                                                                      \
			PW[t] = ROTL(1, PW[(t) - 3] ^ PW[(t) - 8] ^ PW[(t) - 14] ^ PW[(t) - 16]);                      \
			                                                                                               \
		for (int i = 0; i < SMALL_CHUNK_BITMAP_SIZE; ++i)                                                    \
			smallChunkBitmap[i] = CUDA_smallChunkBitmap[i];                                                    \
	}                                                                                                      \
	__syncthreads();                                                                                       \
	randomByte2 += ((threadIdx.x & 0x40) >> 1);                                                            \
	                                                                                                       \
	for (passCount = 0; passCount < CUDA_SHA1_MAX_PASS_COUNT; passCount++){                                \
		__syncthreads();                                                                                   \
		                                                                                                   \
		key2 = tableForKey2[randomByte2 + (passCount >> 6)];                                               \
		key3 = cudaKeyCharTable_SecondByteAndOneByte[randomByte3 + (passCount & 63)];                     \
		                                                                                                   \
		A = H0;                                                                                            \
		B = H1;                                                                                            \
		C = H2;                                                                                            \
		D = H3;                                                                                            \
		E = H4;                                                                                            \
		                                                                                                   \
		unsigned int W0   = (key0 << 24) | (key1 << 16) | (key2 << 8) | key3;                              \
		unsigned int W0_1 = ROTL(1,  W0);                                                                  \
		unsigned int W0_2 = ROTL(2,  W0);                                                                  \
		unsigned int W0_3 = ROTL(3,  W0);                                                                  \
		unsigned int W0_4 = ROTL(4,  W0);                                                                  \
		unsigned int W0_5 = ROTL(5,  W0);                                                                  \
		unsigned int W0_6 = ROTL(6,  W0);                                                                  \
		unsigned int W0_7 = ROTL(7,  W0);                                                                  \
		unsigned int W0_8 = ROTL(8,  W0);                                                                  \
		unsigned int W0_9 = ROTL(9,  W0);                                                                  \
		unsigned int W010 = ROTL(10, W0);                                                                  \
		unsigned int W011 = ROTL(11, W0);                                                                  \
		unsigned int W012 = ROTL(12, W0);                                                                  \
		unsigned int W013 = ROTL(13, W0);                                                                  \
		unsigned int W014 = ROTL(14, W0);                                                                  \
		unsigned int W015 = ROTL(15, W0);                                                                  \
		unsigned int W016 = ROTL(16, W0);                                                                  \
		unsigned int W017 = ROTL(17, W0);                                                                  \
		unsigned int W018 = ROTL(18, W0);                                                                  \
		unsigned int W019 = ROTL(19, W0);                                                                  \
		unsigned int W020 = ROTL(20, W0);                                                                  \
		unsigned int W021 = ROTL(21, W0);                                                                  \
		unsigned int W022 = ROTL(22, W0);                                                                  \
		unsigned int W0_6___W0_4        = W0_6        ^ W0_4;                                              \
		unsigned int W0_6___W0_4___W0_7 = W0_6___W0_4 ^ W0_7;                                              \
		unsigned int W0_8___W0_4        = W0_8        ^ W0_4;                                              \
		unsigned int W0_8___W012        = W0_8        ^ W012;                                              \
		                                                                                                   \
		ROUND_00_TO_19(0,  W0);                                                                            \
		ROUND_00_TO_19(1,  PW[1]);                                                                         \
		ROUND_00_TO_19(2,  PW[2]);                                                                         \
		ROUND_00_TO_19(3,  PW[3]);                                                                         \
		ROUND_00_TO_19(4,  PW[4]);                                                                         \
		ROUND_00_TO_19(5,  PW[5]);                                                                         \
		ROUND_00_TO_19(6,  PW[6]);                                                                         \
		ROUND_00_TO_19(7,  PW[7]);                                                                         \
		ROUND_00_TO_19(8,  PW[8]);                                                                         \
		ROUND_00_TO_19(9,  PW[9]);                                                                         \
		ROUND_00_TO_19(10, PW[10]);                                                                        \
		ROUND_00_TO_19(11, PW[11]);                                                                        \
		ROUND_00_TO_19(12, PW[12]);                                                                        \
		ROUND_00_TO_19(13, PW[13]);                                                                        \
		ROUND_00_TO_19(14, PW[14]);                                                                        \
		ROUND_00_TO_19(15, PW[15]);                                                                        \
		                                                                                                   \
		ROUND_00_TO_19(16, PW[16] ^ W0_1                                   );                              \
		ROUND_00_TO_19(17, PW[17]                                          );                              \
		ROUND_00_TO_19(18, PW[18]                                          );                              \
		ROUND_00_TO_19(19, PW[19] ^ W0_2                                   );                              \
		                                                                                                   \
		ROUND_20_TO_39(20, PW[20]                                          );                              \
		ROUND_20_TO_39(21, PW[21]                                          );                              \
		ROUND_20_TO_39(22, PW[22] ^ W0_3                                   );                              \
		ROUND_20_TO_39(23, PW[23]                                          );                              \
		ROUND_20_TO_39(24, PW[24] ^ W0_2                                   );                              \
		ROUND_20_TO_39(25, PW[25] ^ W0_4                                   );                              \
		ROUND_20_TO_39(26, PW[26]                                          );                              \
		ROUND_20_TO_39(27, PW[27]                                          );                              \
		ROUND_20_TO_39(28, PW[28] ^ W0_5                                   );                              \
		ROUND_20_TO_39(29, PW[29]                                          );                              \
		ROUND_20_TO_39(30, PW[30] ^ W0_4 ^ W0_2                            );                              \
		ROUND_20_TO_39(31, PW[31] ^ W0_6                                   );                              \
		ROUND_20_TO_39(32, PW[32] ^ W0_3 ^ W0_2                            );                              \
		ROUND_20_TO_39(33, PW[33]                                          );                              \
		ROUND_20_TO_39(34, PW[34] ^ W0_7                                   );                              \
		ROUND_20_TO_39(35, PW[35] ^ W0_4                                   );                              \
		ROUND_20_TO_39(36, PW[36] ^ W0_6___W0_4                            );                              \
		ROUND_20_TO_39(37, PW[37] ^ W0_8                                   );                              \
		ROUND_20_TO_39(38, PW[38] ^ W0_4                                   );                              \
		ROUND_20_TO_39(39, PW[39]                                          );                              \
		                                                                                                   \
		ROUND_40_TO_59(40, PW[40] ^ W0_4 ^ W0_9                            );                              \
		ROUND_40_TO_59(41, PW[41]                                          );                              \
		ROUND_40_TO_59(42, PW[42] ^ W0_6 ^ W0_8                            );                              \
		ROUND_40_TO_59(43, PW[43] ^ W010                                   );                              \
		ROUND_40_TO_59(44, PW[44] ^ W0_6 ^ W0_3 ^ W0_7                     );                              \
		ROUND_40_TO_59(45, PW[45]                                          );                              \
		ROUND_40_TO_59(46, PW[46] ^ W0_4 ^ W011                            );                              \
		ROUND_40_TO_59(47, PW[47] ^ W0_8___W0_4                            );                              \
		ROUND_40_TO_59(48, PW[48] ^ W0_8___W0_4 ^ W0_3 ^ W010 ^ W0_5       );                              \
		ROUND_40_TO_59(49, PW[49] ^ W012                                   );                              \
		ROUND_40_TO_59(50, PW[50] ^ W0_8                                   );                              \
		ROUND_40_TO_59(51, PW[51] ^ W0_6___W0_4                            );                              \
		ROUND_40_TO_59(52, PW[52] ^ W0_8___W0_4 ^ W013                     );                              \
		ROUND_40_TO_59(53, PW[53]                                          );                              \
		ROUND_40_TO_59(54, PW[54] ^ W0_7 ^ W010 ^ W012                     );                              \
		ROUND_40_TO_59(55, PW[55] ^ W014                                   );                              \
		ROUND_40_TO_59(56, PW[56] ^ W0_6___W0_4___W0_7 ^ W011 ^ W010       );                              \
		ROUND_40_TO_59(57, PW[57] ^ W0_8                                   );                              \
		ROUND_40_TO_59(58, PW[58] ^ W0_8___W0_4 ^ W015                     );                              \
		ROUND_40_TO_59(59, PW[59] ^ W0_8___W012                            );                              \
		                                                                                                   \
		ROUND_60_TO_79(60, PW[60] ^ W0_8___W012 ^ W0_4 ^ W0_7 ^ W014       );                              \
		ROUND_60_TO_79(61, PW[61] ^ W016                                   );                              \
		ROUND_60_TO_79(62, PW[62] ^ W0_6___W0_4 ^ W0_8___W012              );                              \
		ROUND_60_TO_79(63, PW[63] ^ W0_8                                   );                              \
		ROUND_60_TO_79(64, PW[64] ^ W0_6___W0_4___W0_7 ^ W0_8___W012 ^ W017);                              \
		ROUND_60_TO_79(65, PW[65]                                          );                              \
		ROUND_60_TO_79(66, PW[66] ^ W014 ^ W016                            );                              \
		ROUND_60_TO_79(67, PW[67] ^ W0_8 ^ W018                            );                              \
		ROUND_60_TO_79(68, PW[68] ^ W011 ^ W014 ^ W015                     );                              \
		ROUND_60_TO_79(69, PW[69]                                          );                              \
		ROUND_60_TO_79(70, PW[70] ^ W012 ^ W019                            );                              \
		ROUND_60_TO_79(71, PW[71] ^ W012 ^ W016                            );                              \
		ROUND_60_TO_79(72, PW[72] ^ W011 ^ W012 ^ W018 ^ W013 ^ W016 ^ W0_5);                              \
		ROUND_60_TO_79(73, PW[73] ^ W020                                   );                              \
		ROUND_60_TO_79(74, PW[74] ^ W0_8 ^ W016                            );                              \
		ROUND_60_TO_79(75, PW[75] ^ W0_6 ^ W012 ^ W014                     );                              \
		ROUND_60_TO_79(76, PW[76] ^ W0_7 ^ W0_8 ^ W012 ^ W016 ^ W021       );                              \
		ROUND_60_TO_79(77, PW[77]                                          );                              \
		ROUND_60_TO_79(78, PW[78] ^ W0_7 ^ W0_8 ^ W015 ^ W018 ^ W020       );                              \
		ROUND_60_TO_79(79, PW[79] ^ W0_8 ^ W022                            );                              \
		                                                                                                   \
		A += H0;                                                                                           \
		B += H1;                                                                                           \
		C += H2;                                                                                           \
		                                                                                                   \
		unsigned int tripcodeChunk = A >> 2;                                                               \

#define CUDA_SHA1_USE_SMALL_CHUNK_BITMAP                                                     \
		if (smallChunkBitmap[tripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)]) \
			continue;                                                                      \

#define CUDA_SHA1_USE_CHUNK_BITMAP \
		if (chunkBitmap[tripcodeChunk >> ((5 - CHUNK_BITMAP_LEN_STRING) * 6)]) \
			continue;

#define CUDA_SHA1_LINEAR_SEARCH \
	for (unsigned int i = 0; i < numTripcodeChunk; i++){ \
		if (tripcodeChunkArray[i] == tripcodeChunk) { \
			found = 1; \
			break; \
		} \
	} \
	if (found) \
		break;

#define CUDA_SHA1_BINARY_SEARCH \
		{\
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;         \
			while (tripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) { \
				middle = (lower + upper) >> 1;                                          \
				if (tripcodeChunk > tripcodeChunkArray[middle]) {                   \
					lower = middle + 1;                                                 \
				} else {                                                                \
					upper = middle - 1;                                                 \
				}                                                                       \
			}                                                                           \
			if (tripcodeChunk == tripcodeChunkArray[middle]) {                      \
				found = 1;                                                              \
				break;                                                                  \
			} \
		}

#define CUDA_SHA1_END_OF_SEAERCH_FUNCTION \
	}\
	if (!found) {\
		output->numGeneratedTripcodes = CUDA_SHA1_MAX_PASS_COUNT;  \
	} else {\
		TripcodeKeyPair *pair = &(output->pair);\
		pair->key.c[0]  = key0;\
		pair->key.c[1]  = key1;\
		pair->key.c[2]  = key2;\
		pair->key.c[3]  = key3;\
		pair->key.c[7]  = keyAndRandomBytes[7];\
		pair->key.c[11] = key11;\
		pair->tripcode.c[0]  = CUDA_base64CharTable[ A >> 26                  ];\
		pair->tripcode.c[1]  = CUDA_base64CharTable[(A >> 20          ) & 0x3f];\
		pair->tripcode.c[2]  = CUDA_base64CharTable[(A >> 14          ) & 0x3f];\
		pair->tripcode.c[3]  = CUDA_base64CharTable[(A >>  8          ) & 0x3f];\
		pair->tripcode.c[4]  = CUDA_base64CharTable[(A >>  2          ) & 0x3f];\
		pair->tripcode.c[5]  = CUDA_base64CharTable[(B >> 28 | A <<  4) & 0x3f];\
		pair->tripcode.c[6]  = CUDA_base64CharTable[(B >> 22          ) & 0x3f];\
		pair->tripcode.c[7]  = CUDA_base64CharTable[(B >> 16          ) & 0x3f];\
		pair->tripcode.c[8]  = CUDA_base64CharTable[(B >> 10          ) & 0x3f];\
		pair->tripcode.c[9]  = CUDA_base64CharTable[(B >>  4          ) & 0x3f];\
		pair->tripcode.c[10] = CUDA_base64CharTable[(B <<  2 | C >> 30) & 0x3f];\
		pair->tripcode.c[11] = CUDA_base64CharTable[(C >> 24          ) & 0x3f];\
		output->numMatchingTripcodes = 1;\
		output->numGeneratedTripcodes = passCount + 1;\
	}\
}

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_ForwardMatching_1Chunk)
	unsigned int      tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_SHA1_BEFORE_SEARCHING
	if (tripcodeChunk == tripcodeChunk0) {
		found = 1;
		break;
	}
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_ForwardMatching_Simple)
CUDA_SHA1_BEFORE_SEARCHING
	CUDA_SHA1_USE_SMALL_CHUNK_BITMAP
	CUDA_SHA1_LINEAR_SEARCH
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_ForwardMatching)
CUDA_SHA1_BEFORE_SEARCHING
	CUDA_SHA1_USE_SMALL_CHUNK_BITMAP
	CUDA_SHA1_USE_CHUNK_BITMAP
	CUDA_SHA1_BINARY_SEARCH
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_BackwardMatching_Simple)
CUDA_SHA1_BEFORE_SEARCHING
	tripcodeChunk = ((B <<  8) & 0x3fffffff) | ((C >> 24) & 0x000000ff);
	CUDA_SHA1_USE_SMALL_CHUNK_BITMAP
	CUDA_SHA1_LINEAR_SEARCH
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_BackwardMatching)
CUDA_SHA1_BEFORE_SEARCHING
	tripcodeChunk = ((B <<  8) & 0x3fffffff) | ((C >> 24) & 0x000000ff);
	CUDA_SHA1_USE_SMALL_CHUNK_BITMAP
	CUDA_SHA1_USE_CHUNK_BITMAP
	CUDA_SHA1_BINARY_SEARCH
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_Flexible_Simple)
CUDA_SHA1_BEFORE_SEARCHING

#define PERFORM_LINEAR_SEARCH_IF_NECESSARY                                           \
	if (!smallChunkBitmap[tripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)]) { \
		CUDA_SHA1_LINEAR_SEARCH                                                      \
	}                                                                                \

	/* tripcodeChunk =  (A >>  2) */                                        PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A <<  4) & 0x3fffffff) | ((B >> 28) & 0x0000000f); PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 10) & 0x3fffffff) | ((B >> 22) & 0x000003ff); PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 16) & 0x3fffffff) | ((B >> 16) & 0x0000ffff); PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 22) & 0x3fffffff) | ((B >> 10) & 0x003fffff); PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 28) & 0x3fffffff) | ((B >>  4) & 0x0fffffff); PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((B <<  2) & 0x3fffffff) | ((C >> 30) & 0x00000003); PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((B <<  8) & 0x3fffffff) | ((C >> 24) & 0x000000ff); PERFORM_LINEAR_SEARCH_IF_NECESSARY
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_Flexible)
CUDA_SHA1_BEFORE_SEARCHING

#define PERFORM_BINARY_SEARCH_IF_NECESSARY                                              \
	if (   !smallChunkBitmap[tripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)]    \
	    && !chunkBitmap     [tripcodeChunk >> ((5 - CHUNK_BITMAP_LEN_STRING      ) * 6)]) { \
		CUDA_SHA1_BINARY_SEARCH                                                         \
	}                                                                                   \

	/* tripcodeChunk =  (A >>  2) */                                        PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A <<  4) & 0x3fffffff) | ((B >> 28) & 0x0000000f); PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 10) & 0x3fffffff) | ((B >> 22) & 0x000003ff); PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 16) & 0x3fffffff) | ((B >> 16) & 0x0000ffff); PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 22) & 0x3fffffff) | ((B >> 10) & 0x003fffff); PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((A << 28) & 0x3fffffff) | ((B >>  4) & 0x0fffffff); PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((B <<  2) & 0x3fffffff) | ((C >> 30) & 0x00000003); PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((B <<  8) & 0x3fffffff) | ((C >> 24) & 0x000000ff); PERFORM_BINARY_SEARCH_IF_NECESSARY
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_ForwardAndBackwardMatching_Simple)
CUDA_SHA1_BEFORE_SEARCHING
	/* tripcodeChunk =  (A >>  2) */                                        PERFORM_LINEAR_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((B <<  8) & 0x3fffffff) | ((C >> 24) & 0x000000ff); PERFORM_LINEAR_SEARCH_IF_NECESSARY
CUDA_SHA1_END_OF_SEAERCH_FUNCTION

CUDA_SHA1_DEFINE_SEARCH_FUNCTION(CUDA_SHA1_PerformSearching_ForwardAndBackwardMatching)
CUDA_SHA1_BEFORE_SEARCHING
	/* tripcodeChunk =  (A >>  2) */                                        PERFORM_BINARY_SEARCH_IF_NECESSARY
	   tripcodeChunk = ((B <<  8) & 0x3fffffff) | ((C >> 24) & 0x000000ff); PERFORM_BINARY_SEARCH_IF_NECESSARY
CUDA_SHA1_END_OF_SEAERCH_FUNCTION



///////////////////////////////////////////////////////////////////////////////
// CUDA SEARCH THREAD FOR 12 CHARACTER TRIPCODES                             //
///////////////////////////////////////////////////////////////////////////////

unsigned WINAPI Thread_SearchForSHA1TripcodesOnCUDADevice(LPVOID info)
{
	hipDeviceProp_t CUDADeviceProperties;
	unsigned int         numBlocksPerSM;
	unsigned int         numBlocksPerGrid;
	GPUOutput *outputArray = NULL;
	GPUOutput *CUDA_outputArray = NULL;
	unsigned int     *CUDA_tripcodeChunkArray = NULL;
	unsigned char      *CUDA_chunkBitmap = NULL;
	unsigned char      *cudaKeyAndRandomBytes;
	unsigned int      sizeOutputArray;
	unsigned char       key[MAX_LEN_TRIPCODE + 1];
	char        status[LEN_LINE_BUFFER_FOR_SCREEN] = "";
	double      timeElapsed = 0;
	double      numGeneratedTripcodes = 0;
	double      speed = 0;
	DWORD       startingTime;
	DWORD       endingTime;
	double      deltaTime;

	key[lenTripcode] = '\0';
	
	CUDA_ERROR(hipSetDevice(((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	CUDA_ERROR(hipGetDeviceProperties(&CUDADeviceProperties, ((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	if (CUDADeviceProperties.computeMode == hipComputeModeProhibited) {
		sprintf(status, "[disabled]");
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), status);
		return 0;
	}

	numBlocksPerSM = options.CUDANumBlocksPerSM;
	numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
	sizeOutputArray = CUDA_SHA1_NUM_THREADS_PER_BLOCK * numBlocksPerGrid;
	outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(outputArray == NULL, ERROR_NO_MEMORY, GetErrorMessage(ERROR_NO_MEMORY));
	CUDA_ERROR(hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray));
	CUDA_ERROR(hipMalloc((void **)&CUDA_chunkBitmap, CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMalloc((void **)&CUDA_tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk)); 
	CUDA_ERROR(hipMalloc((void **)&cudaKeyAndRandomBytes, sizeof(unsigned char) * 12)); 
 
	(((CUDADeviceSearchThreadInfo *)info)->criticalSection).lock();
	CUDA_ERROR(hipMemcpy(CUDA_tripcodeChunkArray, tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpy(CUDA_chunkBitmap, chunkBitmap, CHUNK_BITMAP_SIZE, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),                   base64CharTable,                    sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_OneByte),              keyCharTable_OneByte,               SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),            keyCharTable_FirstByte,             SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),           keyCharTable_SecondByte,            SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByteAndOneByte), keyCharTable_SecondByteAndOneByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_smallChunkBitmap),                    smallChunkBitmap,                     SMALL_CHUNK_BITMAP_SIZE));
	(((CUDADeviceSearchThreadInfo *)info)->criticalSection).unlock();

	startingTime = timeGetTime();

	while (!GetTerminationState()) {
		// Choose a random key.
		SetCharactersInTripcodeKeyForSHA1Tripcode(key);
		if (!IsValidKey(key))
			continue;
		for (int i = 0; i < 4; ++i)
			key[i] = RandomByte();
		key[11] = RandomByte();
				
		// Call an appropriate CUDA function.
		CUDA_ERROR(hipMemcpy(cudaKeyAndRandomBytes, key, 12, hipMemcpyHostToDevice));
		dim3 dimBlock(CUDA_SHA1_NUM_THREADS_PER_BLOCK);
		dim3 dimGrid(numBlocksPerGrid);
		if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
			if (numTripcodeChunk == 1) {
				CUDA_SHA1_PerformSearching_ForwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			} else if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_SHA1_PerformSearching_ForwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			} else {
				CUDA_SHA1_PerformSearching_ForwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			}
		
		} else if (searchMode == SEARCH_MODE_BACKWARD_MATCHING) {
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_SHA1_PerformSearching_BackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			} else {
				CUDA_SHA1_PerformSearching_BackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			}

		} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_SHA1_PerformSearching_ForwardAndBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			} else {
				CUDA_SHA1_PerformSearching_ForwardAndBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			}
		} else {
			// Flexible search
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_SHA1_PerformSearching_Flexible_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			} else {
				CUDA_SHA1_PerformSearching_Flexible<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
				    cudaKeyAndRandomBytes);
			}
		}
		CUDA_ERROR(hipGetLastError());

		// Process the output array.
		CUDA_ERROR(hipMemcpy(outputArray, CUDA_outputArray, sizeOutputArray * sizeof(GPUOutput), hipMemcpyDeviceToHost));
		numGeneratedTripcodes += ProcessGPUOutput(key, outputArray, sizeOutputArray, TRUE);

		//
		endingTime = timeGetTime();
		deltaTime = (endingTime >= startingTime)
								? ((double)endingTime - (double)startingTime                     ) * 0.001
								: ((double)endingTime - (double)startingTime + (double)0xffffffff) * 0.001;
		while (GetPauseState() && !GetTerminationState())
			Sleep(PAUSE_INTERVAL);
		startingTime = timeGetTime();
		timeElapsed += deltaTime;
		speed = numGeneratedTripcodes / timeElapsed;
		sprintf(status,
			    "%.1lfM TPS, %d blocks/SM",
				speed / 1000000,
				numBlocksPerSM);
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), status);
	}

	RELEASE_AND_SET_TO_NULL(CUDA_outputArray,        hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_tripcodeChunkArray, hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_chunkBitmap,        hipFree);
	RELEASE_AND_SET_TO_NULL(cudaKeyAndRandomBytes,                 hipFree);
	RELEASE_AND_SET_TO_NULL(outputArray,             free);
}


