// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 2560
#include "CUDA10_Registers_Kernel.h"
#define SALT 2561
#include "CUDA10_Registers_Kernel.h"
#define SALT 2562
#include "CUDA10_Registers_Kernel.h"
#define SALT 2563
#include "CUDA10_Registers_Kernel.h"
#define SALT 2564
#include "CUDA10_Registers_Kernel.h"
#define SALT 2565
#include "CUDA10_Registers_Kernel.h"
#define SALT 2566
#include "CUDA10_Registers_Kernel.h"
#define SALT 2567
#include "CUDA10_Registers_Kernel.h"
#define SALT 2568
#include "CUDA10_Registers_Kernel.h"
#define SALT 2569
#include "CUDA10_Registers_Kernel.h"
#define SALT 2570
#include "CUDA10_Registers_Kernel.h"
#define SALT 2571
#include "CUDA10_Registers_Kernel.h"
#define SALT 2572
#include "CUDA10_Registers_Kernel.h"
#define SALT 2573
#include "CUDA10_Registers_Kernel.h"
#define SALT 2574
#include "CUDA10_Registers_Kernel.h"
#define SALT 2575
#include "CUDA10_Registers_Kernel.h"
#define SALT 2576
#include "CUDA10_Registers_Kernel.h"
#define SALT 2577
#include "CUDA10_Registers_Kernel.h"
#define SALT 2578
#include "CUDA10_Registers_Kernel.h"
#define SALT 2579
#include "CUDA10_Registers_Kernel.h"
#define SALT 2580
#include "CUDA10_Registers_Kernel.h"
#define SALT 2581
#include "CUDA10_Registers_Kernel.h"
#define SALT 2582
#include "CUDA10_Registers_Kernel.h"
#define SALT 2583
#include "CUDA10_Registers_Kernel.h"
#define SALT 2584
#include "CUDA10_Registers_Kernel.h"
#define SALT 2585
#include "CUDA10_Registers_Kernel.h"
#define SALT 2586
#include "CUDA10_Registers_Kernel.h"
#define SALT 2587
#include "CUDA10_Registers_Kernel.h"
#define SALT 2588
#include "CUDA10_Registers_Kernel.h"
#define SALT 2589
#include "CUDA10_Registers_Kernel.h"
#define SALT 2590
#include "CUDA10_Registers_Kernel.h"
#define SALT 2591
#include "CUDA10_Registers_Kernel.h"
#define SALT 2592
#include "CUDA10_Registers_Kernel.h"
#define SALT 2593
#include "CUDA10_Registers_Kernel.h"
#define SALT 2594
#include "CUDA10_Registers_Kernel.h"
#define SALT 2595
#include "CUDA10_Registers_Kernel.h"
#define SALT 2596
#include "CUDA10_Registers_Kernel.h"
#define SALT 2597
#include "CUDA10_Registers_Kernel.h"
#define SALT 2598
#include "CUDA10_Registers_Kernel.h"
#define SALT 2599
#include "CUDA10_Registers_Kernel.h"
#define SALT 2600
#include "CUDA10_Registers_Kernel.h"
#define SALT 2601
#include "CUDA10_Registers_Kernel.h"
#define SALT 2602
#include "CUDA10_Registers_Kernel.h"
#define SALT 2603
#include "CUDA10_Registers_Kernel.h"
#define SALT 2604
#include "CUDA10_Registers_Kernel.h"
#define SALT 2605
#include "CUDA10_Registers_Kernel.h"
#define SALT 2606
#include "CUDA10_Registers_Kernel.h"
#define SALT 2607
#include "CUDA10_Registers_Kernel.h"
#define SALT 2608
#include "CUDA10_Registers_Kernel.h"
#define SALT 2609
#include "CUDA10_Registers_Kernel.h"
#define SALT 2610
#include "CUDA10_Registers_Kernel.h"
#define SALT 2611
#include "CUDA10_Registers_Kernel.h"
#define SALT 2612
#include "CUDA10_Registers_Kernel.h"
#define SALT 2613
#include "CUDA10_Registers_Kernel.h"
#define SALT 2614
#include "CUDA10_Registers_Kernel.h"
#define SALT 2615
#include "CUDA10_Registers_Kernel.h"
#define SALT 2616
#include "CUDA10_Registers_Kernel.h"
#define SALT 2617
#include "CUDA10_Registers_Kernel.h"
#define SALT 2618
#include "CUDA10_Registers_Kernel.h"
#define SALT 2619
#include "CUDA10_Registers_Kernel.h"
#define SALT 2620
#include "CUDA10_Registers_Kernel.h"
#define SALT 2621
#include "CUDA10_Registers_Kernel.h"
#define SALT 2622
#include "CUDA10_Registers_Kernel.h"
#define SALT 2623
#include "CUDA10_Registers_Kernel.h"
#define SALT 2624
#include "CUDA10_Registers_Kernel.h"
#define SALT 2625
#include "CUDA10_Registers_Kernel.h"
#define SALT 2626
#include "CUDA10_Registers_Kernel.h"
#define SALT 2627
#include "CUDA10_Registers_Kernel.h"
#define SALT 2628
#include "CUDA10_Registers_Kernel.h"
#define SALT 2629
#include "CUDA10_Registers_Kernel.h"
#define SALT 2630
#include "CUDA10_Registers_Kernel.h"
#define SALT 2631
#include "CUDA10_Registers_Kernel.h"
#define SALT 2632
#include "CUDA10_Registers_Kernel.h"
#define SALT 2633
#include "CUDA10_Registers_Kernel.h"
#define SALT 2634
#include "CUDA10_Registers_Kernel.h"
#define SALT 2635
#include "CUDA10_Registers_Kernel.h"
#define SALT 2636
#include "CUDA10_Registers_Kernel.h"
#define SALT 2637
#include "CUDA10_Registers_Kernel.h"
#define SALT 2638
#include "CUDA10_Registers_Kernel.h"
#define SALT 2639
#include "CUDA10_Registers_Kernel.h"
#define SALT 2640
#include "CUDA10_Registers_Kernel.h"
#define SALT 2641
#include "CUDA10_Registers_Kernel.h"
#define SALT 2642
#include "CUDA10_Registers_Kernel.h"
#define SALT 2643
#include "CUDA10_Registers_Kernel.h"
#define SALT 2644
#include "CUDA10_Registers_Kernel.h"
#define SALT 2645
#include "CUDA10_Registers_Kernel.h"
#define SALT 2646
#include "CUDA10_Registers_Kernel.h"
#define SALT 2647
#include "CUDA10_Registers_Kernel.h"
#define SALT 2648
#include "CUDA10_Registers_Kernel.h"
#define SALT 2649
#include "CUDA10_Registers_Kernel.h"
#define SALT 2650
#include "CUDA10_Registers_Kernel.h"
#define SALT 2651
#include "CUDA10_Registers_Kernel.h"
#define SALT 2652
#include "CUDA10_Registers_Kernel.h"
#define SALT 2653
#include "CUDA10_Registers_Kernel.h"
#define SALT 2654
#include "CUDA10_Registers_Kernel.h"
#define SALT 2655
#include "CUDA10_Registers_Kernel.h"
#define SALT 2656
#include "CUDA10_Registers_Kernel.h"
#define SALT 2657
#include "CUDA10_Registers_Kernel.h"
#define SALT 2658
#include "CUDA10_Registers_Kernel.h"
#define SALT 2659
#include "CUDA10_Registers_Kernel.h"
#define SALT 2660
#include "CUDA10_Registers_Kernel.h"
#define SALT 2661
#include "CUDA10_Registers_Kernel.h"
#define SALT 2662
#include "CUDA10_Registers_Kernel.h"
#define SALT 2663
#include "CUDA10_Registers_Kernel.h"
#define SALT 2664
#include "CUDA10_Registers_Kernel.h"
#define SALT 2665
#include "CUDA10_Registers_Kernel.h"
#define SALT 2666
#include "CUDA10_Registers_Kernel.h"
#define SALT 2667
#include "CUDA10_Registers_Kernel.h"
#define SALT 2668
#include "CUDA10_Registers_Kernel.h"
#define SALT 2669
#include "CUDA10_Registers_Kernel.h"
#define SALT 2670
#include "CUDA10_Registers_Kernel.h"
#define SALT 2671
#include "CUDA10_Registers_Kernel.h"
#define SALT 2672
#include "CUDA10_Registers_Kernel.h"
#define SALT 2673
#include "CUDA10_Registers_Kernel.h"
#define SALT 2674
#include "CUDA10_Registers_Kernel.h"
#define SALT 2675
#include "CUDA10_Registers_Kernel.h"
#define SALT 2676
#include "CUDA10_Registers_Kernel.h"
#define SALT 2677
#include "CUDA10_Registers_Kernel.h"
#define SALT 2678
#include "CUDA10_Registers_Kernel.h"
#define SALT 2679
#include "CUDA10_Registers_Kernel.h"
#define SALT 2680
#include "CUDA10_Registers_Kernel.h"
#define SALT 2681
#include "CUDA10_Registers_Kernel.h"
#define SALT 2682
#include "CUDA10_Registers_Kernel.h"
#define SALT 2683
#include "CUDA10_Registers_Kernel.h"
#define SALT 2684
#include "CUDA10_Registers_Kernel.h"
#define SALT 2685
#include "CUDA10_Registers_Kernel.h"
#define SALT 2686
#include "CUDA10_Registers_Kernel.h"
#define SALT 2687
#include "CUDA10_Registers_Kernel.h"
#define SALT 2688
#include "CUDA10_Registers_Kernel.h"
#define SALT 2689
#include "CUDA10_Registers_Kernel.h"
#define SALT 2690
#include "CUDA10_Registers_Kernel.h"
#define SALT 2691
#include "CUDA10_Registers_Kernel.h"
#define SALT 2692
#include "CUDA10_Registers_Kernel.h"
#define SALT 2693
#include "CUDA10_Registers_Kernel.h"
#define SALT 2694
#include "CUDA10_Registers_Kernel.h"
#define SALT 2695
#include "CUDA10_Registers_Kernel.h"
#define SALT 2696
#include "CUDA10_Registers_Kernel.h"
#define SALT 2697
#include "CUDA10_Registers_Kernel.h"
#define SALT 2698
#include "CUDA10_Registers_Kernel.h"
#define SALT 2699
#include "CUDA10_Registers_Kernel.h"
#define SALT 2700
#include "CUDA10_Registers_Kernel.h"
#define SALT 2701
#include "CUDA10_Registers_Kernel.h"
#define SALT 2702
#include "CUDA10_Registers_Kernel.h"
#define SALT 2703
#include "CUDA10_Registers_Kernel.h"
#define SALT 2704
#include "CUDA10_Registers_Kernel.h"
#define SALT 2705
#include "CUDA10_Registers_Kernel.h"
#define SALT 2706
#include "CUDA10_Registers_Kernel.h"
#define SALT 2707
#include "CUDA10_Registers_Kernel.h"
#define SALT 2708
#include "CUDA10_Registers_Kernel.h"
#define SALT 2709
#include "CUDA10_Registers_Kernel.h"
#define SALT 2710
#include "CUDA10_Registers_Kernel.h"
#define SALT 2711
#include "CUDA10_Registers_Kernel.h"
#define SALT 2712
#include "CUDA10_Registers_Kernel.h"
#define SALT 2713
#include "CUDA10_Registers_Kernel.h"
#define SALT 2714
#include "CUDA10_Registers_Kernel.h"
#define SALT 2715
#include "CUDA10_Registers_Kernel.h"
#define SALT 2716
#include "CUDA10_Registers_Kernel.h"
#define SALT 2717
#include "CUDA10_Registers_Kernel.h"
#define SALT 2718
#include "CUDA10_Registers_Kernel.h"
#define SALT 2719
#include "CUDA10_Registers_Kernel.h"
#define SALT 2720
#include "CUDA10_Registers_Kernel.h"
#define SALT 2721
#include "CUDA10_Registers_Kernel.h"
#define SALT 2722
#include "CUDA10_Registers_Kernel.h"
#define SALT 2723
#include "CUDA10_Registers_Kernel.h"
#define SALT 2724
#include "CUDA10_Registers_Kernel.h"
#define SALT 2725
#include "CUDA10_Registers_Kernel.h"
#define SALT 2726
#include "CUDA10_Registers_Kernel.h"
#define SALT 2727
#include "CUDA10_Registers_Kernel.h"
#define SALT 2728
#include "CUDA10_Registers_Kernel.h"
#define SALT 2729
#include "CUDA10_Registers_Kernel.h"
#define SALT 2730
#include "CUDA10_Registers_Kernel.h"
#define SALT 2731
#include "CUDA10_Registers_Kernel.h"
#define SALT 2732
#include "CUDA10_Registers_Kernel.h"
#define SALT 2733
#include "CUDA10_Registers_Kernel.h"
#define SALT 2734
#include "CUDA10_Registers_Kernel.h"
#define SALT 2735
#include "CUDA10_Registers_Kernel.h"
#define SALT 2736
#include "CUDA10_Registers_Kernel.h"
#define SALT 2737
#include "CUDA10_Registers_Kernel.h"
#define SALT 2738
#include "CUDA10_Registers_Kernel.h"
#define SALT 2739
#include "CUDA10_Registers_Kernel.h"
#define SALT 2740
#include "CUDA10_Registers_Kernel.h"
#define SALT 2741
#include "CUDA10_Registers_Kernel.h"
#define SALT 2742
#include "CUDA10_Registers_Kernel.h"
#define SALT 2743
#include "CUDA10_Registers_Kernel.h"
#define SALT 2744
#include "CUDA10_Registers_Kernel.h"
#define SALT 2745
#include "CUDA10_Registers_Kernel.h"
#define SALT 2746
#include "CUDA10_Registers_Kernel.h"
#define SALT 2747
#include "CUDA10_Registers_Kernel.h"
#define SALT 2748
#include "CUDA10_Registers_Kernel.h"
#define SALT 2749
#include "CUDA10_Registers_Kernel.h"
#define SALT 2750
#include "CUDA10_Registers_Kernel.h"
#define SALT 2751
#include "CUDA10_Registers_Kernel.h"
#define SALT 2752
#include "CUDA10_Registers_Kernel.h"
#define SALT 2753
#include "CUDA10_Registers_Kernel.h"
#define SALT 2754
#include "CUDA10_Registers_Kernel.h"
#define SALT 2755
#include "CUDA10_Registers_Kernel.h"
#define SALT 2756
#include "CUDA10_Registers_Kernel.h"
#define SALT 2757
#include "CUDA10_Registers_Kernel.h"
#define SALT 2758
#include "CUDA10_Registers_Kernel.h"
#define SALT 2759
#include "CUDA10_Registers_Kernel.h"
#define SALT 2760
#include "CUDA10_Registers_Kernel.h"
#define SALT 2761
#include "CUDA10_Registers_Kernel.h"
#define SALT 2762
#include "CUDA10_Registers_Kernel.h"
#define SALT 2763
#include "CUDA10_Registers_Kernel.h"
#define SALT 2764
#include "CUDA10_Registers_Kernel.h"
#define SALT 2765
#include "CUDA10_Registers_Kernel.h"
#define SALT 2766
#include "CUDA10_Registers_Kernel.h"
#define SALT 2767
#include "CUDA10_Registers_Kernel.h"
#define SALT 2768
#include "CUDA10_Registers_Kernel.h"
#define SALT 2769
#include "CUDA10_Registers_Kernel.h"
#define SALT 2770
#include "CUDA10_Registers_Kernel.h"
#define SALT 2771
#include "CUDA10_Registers_Kernel.h"
#define SALT 2772
#include "CUDA10_Registers_Kernel.h"
#define SALT 2773
#include "CUDA10_Registers_Kernel.h"
#define SALT 2774
#include "CUDA10_Registers_Kernel.h"
#define SALT 2775
#include "CUDA10_Registers_Kernel.h"
#define SALT 2776
#include "CUDA10_Registers_Kernel.h"
#define SALT 2777
#include "CUDA10_Registers_Kernel.h"
#define SALT 2778
#include "CUDA10_Registers_Kernel.h"
#define SALT 2779
#include "CUDA10_Registers_Kernel.h"
#define SALT 2780
#include "CUDA10_Registers_Kernel.h"
#define SALT 2781
#include "CUDA10_Registers_Kernel.h"
#define SALT 2782
#include "CUDA10_Registers_Kernel.h"
#define SALT 2783
#include "CUDA10_Registers_Kernel.h"
#define SALT 2784
#include "CUDA10_Registers_Kernel.h"
#define SALT 2785
#include "CUDA10_Registers_Kernel.h"
#define SALT 2786
#include "CUDA10_Registers_Kernel.h"
#define SALT 2787
#include "CUDA10_Registers_Kernel.h"
#define SALT 2788
#include "CUDA10_Registers_Kernel.h"
#define SALT 2789
#include "CUDA10_Registers_Kernel.h"
#define SALT 2790
#include "CUDA10_Registers_Kernel.h"
#define SALT 2791
#include "CUDA10_Registers_Kernel.h"
#define SALT 2792
#include "CUDA10_Registers_Kernel.h"
#define SALT 2793
#include "CUDA10_Registers_Kernel.h"
#define SALT 2794
#include "CUDA10_Registers_Kernel.h"
#define SALT 2795
#include "CUDA10_Registers_Kernel.h"
#define SALT 2796
#include "CUDA10_Registers_Kernel.h"
#define SALT 2797
#include "CUDA10_Registers_Kernel.h"
#define SALT 2798
#include "CUDA10_Registers_Kernel.h"
#define SALT 2799
#include "CUDA10_Registers_Kernel.h"
#define SALT 2800
#include "CUDA10_Registers_Kernel.h"
#define SALT 2801
#include "CUDA10_Registers_Kernel.h"
#define SALT 2802
#include "CUDA10_Registers_Kernel.h"
#define SALT 2803
#include "CUDA10_Registers_Kernel.h"
#define SALT 2804
#include "CUDA10_Registers_Kernel.h"
#define SALT 2805
#include "CUDA10_Registers_Kernel.h"
#define SALT 2806
#include "CUDA10_Registers_Kernel.h"
#define SALT 2807
#include "CUDA10_Registers_Kernel.h"
#define SALT 2808
#include "CUDA10_Registers_Kernel.h"
#define SALT 2809
#include "CUDA10_Registers_Kernel.h"
#define SALT 2810
#include "CUDA10_Registers_Kernel.h"
#define SALT 2811
#include "CUDA10_Registers_Kernel.h"
#define SALT 2812
#include "CUDA10_Registers_Kernel.h"
#define SALT 2813
#include "CUDA10_Registers_Kernel.h"
#define SALT 2814
#include "CUDA10_Registers_Kernel.h"
#define SALT 2815
#include "CUDA10_Registers_Kernel.h"
#define SALT 2816
#include "CUDA10_Registers_Kernel.h"
#define SALT 2817
#include "CUDA10_Registers_Kernel.h"
#define SALT 2818
#include "CUDA10_Registers_Kernel.h"
#define SALT 2819
#include "CUDA10_Registers_Kernel.h"
#define SALT 2820
#include "CUDA10_Registers_Kernel.h"
#define SALT 2821
#include "CUDA10_Registers_Kernel.h"
#define SALT 2822
#include "CUDA10_Registers_Kernel.h"
#define SALT 2823
#include "CUDA10_Registers_Kernel.h"
#define SALT 2824
#include "CUDA10_Registers_Kernel.h"
#define SALT 2825
#include "CUDA10_Registers_Kernel.h"
#define SALT 2826
#include "CUDA10_Registers_Kernel.h"
#define SALT 2827
#include "CUDA10_Registers_Kernel.h"
#define SALT 2828
#include "CUDA10_Registers_Kernel.h"
#define SALT 2829
#include "CUDA10_Registers_Kernel.h"
#define SALT 2830
#include "CUDA10_Registers_Kernel.h"
#define SALT 2831
#include "CUDA10_Registers_Kernel.h"
#define SALT 2832
#include "CUDA10_Registers_Kernel.h"
#define SALT 2833
#include "CUDA10_Registers_Kernel.h"
#define SALT 2834
#include "CUDA10_Registers_Kernel.h"
#define SALT 2835
#include "CUDA10_Registers_Kernel.h"
#define SALT 2836
#include "CUDA10_Registers_Kernel.h"
#define SALT 2837
#include "CUDA10_Registers_Kernel.h"
#define SALT 2838
#include "CUDA10_Registers_Kernel.h"
#define SALT 2839
#include "CUDA10_Registers_Kernel.h"
#define SALT 2840
#include "CUDA10_Registers_Kernel.h"
#define SALT 2841
#include "CUDA10_Registers_Kernel.h"
#define SALT 2842
#include "CUDA10_Registers_Kernel.h"
#define SALT 2843
#include "CUDA10_Registers_Kernel.h"
#define SALT 2844
#include "CUDA10_Registers_Kernel.h"
#define SALT 2845
#include "CUDA10_Registers_Kernel.h"
#define SALT 2846
#include "CUDA10_Registers_Kernel.h"
#define SALT 2847
#include "CUDA10_Registers_Kernel.h"
#define SALT 2848
#include "CUDA10_Registers_Kernel.h"
#define SALT 2849
#include "CUDA10_Registers_Kernel.h"
#define SALT 2850
#include "CUDA10_Registers_Kernel.h"
#define SALT 2851
#include "CUDA10_Registers_Kernel.h"
#define SALT 2852
#include "CUDA10_Registers_Kernel.h"
#define SALT 2853
#include "CUDA10_Registers_Kernel.h"
#define SALT 2854
#include "CUDA10_Registers_Kernel.h"
#define SALT 2855
#include "CUDA10_Registers_Kernel.h"
#define SALT 2856
#include "CUDA10_Registers_Kernel.h"
#define SALT 2857
#include "CUDA10_Registers_Kernel.h"
#define SALT 2858
#include "CUDA10_Registers_Kernel.h"
#define SALT 2859
#include "CUDA10_Registers_Kernel.h"
#define SALT 2860
#include "CUDA10_Registers_Kernel.h"
#define SALT 2861
#include "CUDA10_Registers_Kernel.h"
#define SALT 2862
#include "CUDA10_Registers_Kernel.h"
#define SALT 2863
#include "CUDA10_Registers_Kernel.h"
#define SALT 2864
#include "CUDA10_Registers_Kernel.h"
#define SALT 2865
#include "CUDA10_Registers_Kernel.h"
#define SALT 2866
#include "CUDA10_Registers_Kernel.h"
#define SALT 2867
#include "CUDA10_Registers_Kernel.h"
#define SALT 2868
#include "CUDA10_Registers_Kernel.h"
#define SALT 2869
#include "CUDA10_Registers_Kernel.h"
#define SALT 2870
#include "CUDA10_Registers_Kernel.h"
#define SALT 2871
#include "CUDA10_Registers_Kernel.h"
#define SALT 2872
#include "CUDA10_Registers_Kernel.h"
#define SALT 2873
#include "CUDA10_Registers_Kernel.h"
#define SALT 2874
#include "CUDA10_Registers_Kernel.h"
#define SALT 2875
#include "CUDA10_Registers_Kernel.h"
#define SALT 2876
#include "CUDA10_Registers_Kernel.h"
#define SALT 2877
#include "CUDA10_Registers_Kernel.h"
#define SALT 2878
#include "CUDA10_Registers_Kernel.h"
#define SALT 2879
#include "CUDA10_Registers_Kernel.h"
#define SALT 2880
#include "CUDA10_Registers_Kernel.h"
#define SALT 2881
#include "CUDA10_Registers_Kernel.h"
#define SALT 2882
#include "CUDA10_Registers_Kernel.h"
#define SALT 2883
#include "CUDA10_Registers_Kernel.h"
#define SALT 2884
#include "CUDA10_Registers_Kernel.h"
#define SALT 2885
#include "CUDA10_Registers_Kernel.h"
#define SALT 2886
#include "CUDA10_Registers_Kernel.h"
#define SALT 2887
#include "CUDA10_Registers_Kernel.h"
#define SALT 2888
#include "CUDA10_Registers_Kernel.h"
#define SALT 2889
#include "CUDA10_Registers_Kernel.h"
#define SALT 2890
#include "CUDA10_Registers_Kernel.h"
#define SALT 2891
#include "CUDA10_Registers_Kernel.h"
#define SALT 2892
#include "CUDA10_Registers_Kernel.h"
#define SALT 2893
#include "CUDA10_Registers_Kernel.h"
#define SALT 2894
#include "CUDA10_Registers_Kernel.h"
#define SALT 2895
#include "CUDA10_Registers_Kernel.h"
#define SALT 2896
#include "CUDA10_Registers_Kernel.h"
#define SALT 2897
#include "CUDA10_Registers_Kernel.h"
#define SALT 2898
#include "CUDA10_Registers_Kernel.h"
#define SALT 2899
#include "CUDA10_Registers_Kernel.h"
#define SALT 2900
#include "CUDA10_Registers_Kernel.h"
#define SALT 2901
#include "CUDA10_Registers_Kernel.h"
#define SALT 2902
#include "CUDA10_Registers_Kernel.h"
#define SALT 2903
#include "CUDA10_Registers_Kernel.h"
#define SALT 2904
#include "CUDA10_Registers_Kernel.h"
#define SALT 2905
#include "CUDA10_Registers_Kernel.h"
#define SALT 2906
#include "CUDA10_Registers_Kernel.h"
#define SALT 2907
#include "CUDA10_Registers_Kernel.h"
#define SALT 2908
#include "CUDA10_Registers_Kernel.h"
#define SALT 2909
#include "CUDA10_Registers_Kernel.h"
#define SALT 2910
#include "CUDA10_Registers_Kernel.h"
#define SALT 2911
#include "CUDA10_Registers_Kernel.h"
#define SALT 2912
#include "CUDA10_Registers_Kernel.h"
#define SALT 2913
#include "CUDA10_Registers_Kernel.h"
#define SALT 2914
#include "CUDA10_Registers_Kernel.h"
#define SALT 2915
#include "CUDA10_Registers_Kernel.h"
#define SALT 2916
#include "CUDA10_Registers_Kernel.h"
#define SALT 2917
#include "CUDA10_Registers_Kernel.h"
#define SALT 2918
#include "CUDA10_Registers_Kernel.h"
#define SALT 2919
#include "CUDA10_Registers_Kernel.h"
#define SALT 2920
#include "CUDA10_Registers_Kernel.h"
#define SALT 2921
#include "CUDA10_Registers_Kernel.h"
#define SALT 2922
#include "CUDA10_Registers_Kernel.h"
#define SALT 2923
#include "CUDA10_Registers_Kernel.h"
#define SALT 2924
#include "CUDA10_Registers_Kernel.h"
#define SALT 2925
#include "CUDA10_Registers_Kernel.h"
#define SALT 2926
#include "CUDA10_Registers_Kernel.h"
#define SALT 2927
#include "CUDA10_Registers_Kernel.h"
#define SALT 2928
#include "CUDA10_Registers_Kernel.h"
#define SALT 2929
#include "CUDA10_Registers_Kernel.h"
#define SALT 2930
#include "CUDA10_Registers_Kernel.h"
#define SALT 2931
#include "CUDA10_Registers_Kernel.h"
#define SALT 2932
#include "CUDA10_Registers_Kernel.h"
#define SALT 2933
#include "CUDA10_Registers_Kernel.h"
#define SALT 2934
#include "CUDA10_Registers_Kernel.h"
#define SALT 2935
#include "CUDA10_Registers_Kernel.h"
#define SALT 2936
#include "CUDA10_Registers_Kernel.h"
#define SALT 2937
#include "CUDA10_Registers_Kernel.h"
#define SALT 2938
#include "CUDA10_Registers_Kernel.h"
#define SALT 2939
#include "CUDA10_Registers_Kernel.h"
#define SALT 2940
#include "CUDA10_Registers_Kernel.h"
#define SALT 2941
#include "CUDA10_Registers_Kernel.h"
#define SALT 2942
#include "CUDA10_Registers_Kernel.h"
#define SALT 2943
#include "CUDA10_Registers_Kernel.h"
#define SALT 2944
#include "CUDA10_Registers_Kernel.h"
#define SALT 2945
#include "CUDA10_Registers_Kernel.h"
#define SALT 2946
#include "CUDA10_Registers_Kernel.h"
#define SALT 2947
#include "CUDA10_Registers_Kernel.h"
#define SALT 2948
#include "CUDA10_Registers_Kernel.h"
#define SALT 2949
#include "CUDA10_Registers_Kernel.h"
#define SALT 2950
#include "CUDA10_Registers_Kernel.h"
#define SALT 2951
#include "CUDA10_Registers_Kernel.h"
#define SALT 2952
#include "CUDA10_Registers_Kernel.h"
#define SALT 2953
#include "CUDA10_Registers_Kernel.h"
#define SALT 2954
#include "CUDA10_Registers_Kernel.h"
#define SALT 2955
#include "CUDA10_Registers_Kernel.h"
#define SALT 2956
#include "CUDA10_Registers_Kernel.h"
#define SALT 2957
#include "CUDA10_Registers_Kernel.h"
#define SALT 2958
#include "CUDA10_Registers_Kernel.h"
#define SALT 2959
#include "CUDA10_Registers_Kernel.h"
#define SALT 2960
#include "CUDA10_Registers_Kernel.h"
#define SALT 2961
#include "CUDA10_Registers_Kernel.h"
#define SALT 2962
#include "CUDA10_Registers_Kernel.h"
#define SALT 2963
#include "CUDA10_Registers_Kernel.h"
#define SALT 2964
#include "CUDA10_Registers_Kernel.h"
#define SALT 2965
#include "CUDA10_Registers_Kernel.h"
#define SALT 2966
#include "CUDA10_Registers_Kernel.h"
#define SALT 2967
#include "CUDA10_Registers_Kernel.h"
#define SALT 2968
#include "CUDA10_Registers_Kernel.h"
#define SALT 2969
#include "CUDA10_Registers_Kernel.h"
#define SALT 2970
#include "CUDA10_Registers_Kernel.h"
#define SALT 2971
#include "CUDA10_Registers_Kernel.h"
#define SALT 2972
#include "CUDA10_Registers_Kernel.h"
#define SALT 2973
#include "CUDA10_Registers_Kernel.h"
#define SALT 2974
#include "CUDA10_Registers_Kernel.h"
#define SALT 2975
#include "CUDA10_Registers_Kernel.h"
#define SALT 2976
#include "CUDA10_Registers_Kernel.h"
#define SALT 2977
#include "CUDA10_Registers_Kernel.h"
#define SALT 2978
#include "CUDA10_Registers_Kernel.h"
#define SALT 2979
#include "CUDA10_Registers_Kernel.h"
#define SALT 2980
#include "CUDA10_Registers_Kernel.h"
#define SALT 2981
#include "CUDA10_Registers_Kernel.h"
#define SALT 2982
#include "CUDA10_Registers_Kernel.h"
#define SALT 2983
#include "CUDA10_Registers_Kernel.h"
#define SALT 2984
#include "CUDA10_Registers_Kernel.h"
#define SALT 2985
#include "CUDA10_Registers_Kernel.h"
#define SALT 2986
#include "CUDA10_Registers_Kernel.h"
#define SALT 2987
#include "CUDA10_Registers_Kernel.h"
#define SALT 2988
#include "CUDA10_Registers_Kernel.h"
#define SALT 2989
#include "CUDA10_Registers_Kernel.h"
#define SALT 2990
#include "CUDA10_Registers_Kernel.h"
#define SALT 2991
#include "CUDA10_Registers_Kernel.h"
#define SALT 2992
#include "CUDA10_Registers_Kernel.h"
#define SALT 2993
#include "CUDA10_Registers_Kernel.h"
#define SALT 2994
#include "CUDA10_Registers_Kernel.h"
#define SALT 2995
#include "CUDA10_Registers_Kernel.h"
#define SALT 2996
#include "CUDA10_Registers_Kernel.h"
#define SALT 2997
#include "CUDA10_Registers_Kernel.h"
#define SALT 2998
#include "CUDA10_Registers_Kernel.h"
#define SALT 2999
#include "CUDA10_Registers_Kernel.h"
#define SALT 3000
#include "CUDA10_Registers_Kernel.h"
#define SALT 3001
#include "CUDA10_Registers_Kernel.h"
#define SALT 3002
#include "CUDA10_Registers_Kernel.h"
#define SALT 3003
#include "CUDA10_Registers_Kernel.h"
#define SALT 3004
#include "CUDA10_Registers_Kernel.h"
#define SALT 3005
#include "CUDA10_Registers_Kernel.h"
#define SALT 3006
#include "CUDA10_Registers_Kernel.h"
#define SALT 3007
#include "CUDA10_Registers_Kernel.h"
#define SALT 3008
#include "CUDA10_Registers_Kernel.h"
#define SALT 3009
#include "CUDA10_Registers_Kernel.h"
#define SALT 3010
#include "CUDA10_Registers_Kernel.h"
#define SALT 3011
#include "CUDA10_Registers_Kernel.h"
#define SALT 3012
#include "CUDA10_Registers_Kernel.h"
#define SALT 3013
#include "CUDA10_Registers_Kernel.h"
#define SALT 3014
#include "CUDA10_Registers_Kernel.h"
#define SALT 3015
#include "CUDA10_Registers_Kernel.h"
#define SALT 3016
#include "CUDA10_Registers_Kernel.h"
#define SALT 3017
#include "CUDA10_Registers_Kernel.h"
#define SALT 3018
#include "CUDA10_Registers_Kernel.h"
#define SALT 3019
#include "CUDA10_Registers_Kernel.h"
#define SALT 3020
#include "CUDA10_Registers_Kernel.h"
#define SALT 3021
#include "CUDA10_Registers_Kernel.h"
#define SALT 3022
#include "CUDA10_Registers_Kernel.h"
#define SALT 3023
#include "CUDA10_Registers_Kernel.h"
#define SALT 3024
#include "CUDA10_Registers_Kernel.h"
#define SALT 3025
#include "CUDA10_Registers_Kernel.h"
#define SALT 3026
#include "CUDA10_Registers_Kernel.h"
#define SALT 3027
#include "CUDA10_Registers_Kernel.h"
#define SALT 3028
#include "CUDA10_Registers_Kernel.h"
#define SALT 3029
#include "CUDA10_Registers_Kernel.h"
#define SALT 3030
#include "CUDA10_Registers_Kernel.h"
#define SALT 3031
#include "CUDA10_Registers_Kernel.h"
#define SALT 3032
#include "CUDA10_Registers_Kernel.h"
#define SALT 3033
#include "CUDA10_Registers_Kernel.h"
#define SALT 3034
#include "CUDA10_Registers_Kernel.h"
#define SALT 3035
#include "CUDA10_Registers_Kernel.h"
#define SALT 3036
#include "CUDA10_Registers_Kernel.h"
#define SALT 3037
#include "CUDA10_Registers_Kernel.h"
#define SALT 3038
#include "CUDA10_Registers_Kernel.h"
#define SALT 3039
#include "CUDA10_Registers_Kernel.h"
#define SALT 3040
#include "CUDA10_Registers_Kernel.h"
#define SALT 3041
#include "CUDA10_Registers_Kernel.h"
#define SALT 3042
#include "CUDA10_Registers_Kernel.h"
#define SALT 3043
#include "CUDA10_Registers_Kernel.h"
#define SALT 3044
#include "CUDA10_Registers_Kernel.h"
#define SALT 3045
#include "CUDA10_Registers_Kernel.h"
#define SALT 3046
#include "CUDA10_Registers_Kernel.h"
#define SALT 3047
#include "CUDA10_Registers_Kernel.h"
#define SALT 3048
#include "CUDA10_Registers_Kernel.h"
#define SALT 3049
#include "CUDA10_Registers_Kernel.h"
#define SALT 3050
#include "CUDA10_Registers_Kernel.h"
#define SALT 3051
#include "CUDA10_Registers_Kernel.h"
#define SALT 3052
#include "CUDA10_Registers_Kernel.h"
#define SALT 3053
#include "CUDA10_Registers_Kernel.h"
#define SALT 3054
#include "CUDA10_Registers_Kernel.h"
#define SALT 3055
#include "CUDA10_Registers_Kernel.h"
#define SALT 3056
#include "CUDA10_Registers_Kernel.h"
#define SALT 3057
#include "CUDA10_Registers_Kernel.h"
#define SALT 3058
#include "CUDA10_Registers_Kernel.h"
#define SALT 3059
#include "CUDA10_Registers_Kernel.h"
#define SALT 3060
#include "CUDA10_Registers_Kernel.h"
#define SALT 3061
#include "CUDA10_Registers_Kernel.h"
#define SALT 3062
#include "CUDA10_Registers_Kernel.h"
#define SALT 3063
#include "CUDA10_Registers_Kernel.h"
#define SALT 3064
#include "CUDA10_Registers_Kernel.h"
#define SALT 3065
#include "CUDA10_Registers_Kernel.h"
#define SALT 3066
#include "CUDA10_Registers_Kernel.h"
#define SALT 3067
#include "CUDA10_Registers_Kernel.h"
#define SALT 3068
#include "CUDA10_Registers_Kernel.h"
#define SALT 3069
#include "CUDA10_Registers_Kernel.h"
#define SALT 3070
#include "CUDA10_Registers_Kernel.h"
#define SALT 3071
#include "CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher5()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel5(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 2560: LAUNCH_KERNEL(2560); break;
	case 2561: LAUNCH_KERNEL(2561); break;
	case 2562: LAUNCH_KERNEL(2562); break;
	case 2563: LAUNCH_KERNEL(2563); break;
	case 2564: LAUNCH_KERNEL(2564); break;
	case 2565: LAUNCH_KERNEL(2565); break;
	case 2566: LAUNCH_KERNEL(2566); break;
	case 2567: LAUNCH_KERNEL(2567); break;
	case 2568: LAUNCH_KERNEL(2568); break;
	case 2569: LAUNCH_KERNEL(2569); break;
	case 2570: LAUNCH_KERNEL(2570); break;
	case 2571: LAUNCH_KERNEL(2571); break;
	case 2572: LAUNCH_KERNEL(2572); break;
	case 2573: LAUNCH_KERNEL(2573); break;
	case 2574: LAUNCH_KERNEL(2574); break;
	case 2575: LAUNCH_KERNEL(2575); break;
	case 2576: LAUNCH_KERNEL(2576); break;
	case 2577: LAUNCH_KERNEL(2577); break;
	case 2578: LAUNCH_KERNEL(2578); break;
	case 2579: LAUNCH_KERNEL(2579); break;
	case 2580: LAUNCH_KERNEL(2580); break;
	case 2581: LAUNCH_KERNEL(2581); break;
	case 2582: LAUNCH_KERNEL(2582); break;
	case 2583: LAUNCH_KERNEL(2583); break;
	case 2584: LAUNCH_KERNEL(2584); break;
	case 2585: LAUNCH_KERNEL(2585); break;
	case 2586: LAUNCH_KERNEL(2586); break;
	case 2587: LAUNCH_KERNEL(2587); break;
	case 2588: LAUNCH_KERNEL(2588); break;
	case 2589: LAUNCH_KERNEL(2589); break;
	case 2590: LAUNCH_KERNEL(2590); break;
	case 2591: LAUNCH_KERNEL(2591); break;
	case 2592: LAUNCH_KERNEL(2592); break;
	case 2593: LAUNCH_KERNEL(2593); break;
	case 2594: LAUNCH_KERNEL(2594); break;
	case 2595: LAUNCH_KERNEL(2595); break;
	case 2596: LAUNCH_KERNEL(2596); break;
	case 2597: LAUNCH_KERNEL(2597); break;
	case 2598: LAUNCH_KERNEL(2598); break;
	case 2599: LAUNCH_KERNEL(2599); break;
	case 2600: LAUNCH_KERNEL(2600); break;
	case 2601: LAUNCH_KERNEL(2601); break;
	case 2602: LAUNCH_KERNEL(2602); break;
	case 2603: LAUNCH_KERNEL(2603); break;
	case 2604: LAUNCH_KERNEL(2604); break;
	case 2605: LAUNCH_KERNEL(2605); break;
	case 2606: LAUNCH_KERNEL(2606); break;
	case 2607: LAUNCH_KERNEL(2607); break;
	case 2608: LAUNCH_KERNEL(2608); break;
	case 2609: LAUNCH_KERNEL(2609); break;
	case 2610: LAUNCH_KERNEL(2610); break;
	case 2611: LAUNCH_KERNEL(2611); break;
	case 2612: LAUNCH_KERNEL(2612); break;
	case 2613: LAUNCH_KERNEL(2613); break;
	case 2614: LAUNCH_KERNEL(2614); break;
	case 2615: LAUNCH_KERNEL(2615); break;
	case 2616: LAUNCH_KERNEL(2616); break;
	case 2617: LAUNCH_KERNEL(2617); break;
	case 2618: LAUNCH_KERNEL(2618); break;
	case 2619: LAUNCH_KERNEL(2619); break;
	case 2620: LAUNCH_KERNEL(2620); break;
	case 2621: LAUNCH_KERNEL(2621); break;
	case 2622: LAUNCH_KERNEL(2622); break;
	case 2623: LAUNCH_KERNEL(2623); break;
	case 2624: LAUNCH_KERNEL(2624); break;
	case 2625: LAUNCH_KERNEL(2625); break;
	case 2626: LAUNCH_KERNEL(2626); break;
	case 2627: LAUNCH_KERNEL(2627); break;
	case 2628: LAUNCH_KERNEL(2628); break;
	case 2629: LAUNCH_KERNEL(2629); break;
	case 2630: LAUNCH_KERNEL(2630); break;
	case 2631: LAUNCH_KERNEL(2631); break;
	case 2632: LAUNCH_KERNEL(2632); break;
	case 2633: LAUNCH_KERNEL(2633); break;
	case 2634: LAUNCH_KERNEL(2634); break;
	case 2635: LAUNCH_KERNEL(2635); break;
	case 2636: LAUNCH_KERNEL(2636); break;
	case 2637: LAUNCH_KERNEL(2637); break;
	case 2638: LAUNCH_KERNEL(2638); break;
	case 2639: LAUNCH_KERNEL(2639); break;
	case 2640: LAUNCH_KERNEL(2640); break;
	case 2641: LAUNCH_KERNEL(2641); break;
	case 2642: LAUNCH_KERNEL(2642); break;
	case 2643: LAUNCH_KERNEL(2643); break;
	case 2644: LAUNCH_KERNEL(2644); break;
	case 2645: LAUNCH_KERNEL(2645); break;
	case 2646: LAUNCH_KERNEL(2646); break;
	case 2647: LAUNCH_KERNEL(2647); break;
	case 2648: LAUNCH_KERNEL(2648); break;
	case 2649: LAUNCH_KERNEL(2649); break;
	case 2650: LAUNCH_KERNEL(2650); break;
	case 2651: LAUNCH_KERNEL(2651); break;
	case 2652: LAUNCH_KERNEL(2652); break;
	case 2653: LAUNCH_KERNEL(2653); break;
	case 2654: LAUNCH_KERNEL(2654); break;
	case 2655: LAUNCH_KERNEL(2655); break;
	case 2656: LAUNCH_KERNEL(2656); break;
	case 2657: LAUNCH_KERNEL(2657); break;
	case 2658: LAUNCH_KERNEL(2658); break;
	case 2659: LAUNCH_KERNEL(2659); break;
	case 2660: LAUNCH_KERNEL(2660); break;
	case 2661: LAUNCH_KERNEL(2661); break;
	case 2662: LAUNCH_KERNEL(2662); break;
	case 2663: LAUNCH_KERNEL(2663); break;
	case 2664: LAUNCH_KERNEL(2664); break;
	case 2665: LAUNCH_KERNEL(2665); break;
	case 2666: LAUNCH_KERNEL(2666); break;
	case 2667: LAUNCH_KERNEL(2667); break;
	case 2668: LAUNCH_KERNEL(2668); break;
	case 2669: LAUNCH_KERNEL(2669); break;
	case 2670: LAUNCH_KERNEL(2670); break;
	case 2671: LAUNCH_KERNEL(2671); break;
	case 2672: LAUNCH_KERNEL(2672); break;
	case 2673: LAUNCH_KERNEL(2673); break;
	case 2674: LAUNCH_KERNEL(2674); break;
	case 2675: LAUNCH_KERNEL(2675); break;
	case 2676: LAUNCH_KERNEL(2676); break;
	case 2677: LAUNCH_KERNEL(2677); break;
	case 2678: LAUNCH_KERNEL(2678); break;
	case 2679: LAUNCH_KERNEL(2679); break;
	case 2680: LAUNCH_KERNEL(2680); break;
	case 2681: LAUNCH_KERNEL(2681); break;
	case 2682: LAUNCH_KERNEL(2682); break;
	case 2683: LAUNCH_KERNEL(2683); break;
	case 2684: LAUNCH_KERNEL(2684); break;
	case 2685: LAUNCH_KERNEL(2685); break;
	case 2686: LAUNCH_KERNEL(2686); break;
	case 2687: LAUNCH_KERNEL(2687); break;
	case 2688: LAUNCH_KERNEL(2688); break;
	case 2689: LAUNCH_KERNEL(2689); break;
	case 2690: LAUNCH_KERNEL(2690); break;
	case 2691: LAUNCH_KERNEL(2691); break;
	case 2692: LAUNCH_KERNEL(2692); break;
	case 2693: LAUNCH_KERNEL(2693); break;
	case 2694: LAUNCH_KERNEL(2694); break;
	case 2695: LAUNCH_KERNEL(2695); break;
	case 2696: LAUNCH_KERNEL(2696); break;
	case 2697: LAUNCH_KERNEL(2697); break;
	case 2698: LAUNCH_KERNEL(2698); break;
	case 2699: LAUNCH_KERNEL(2699); break;
	case 2700: LAUNCH_KERNEL(2700); break;
	case 2701: LAUNCH_KERNEL(2701); break;
	case 2702: LAUNCH_KERNEL(2702); break;
	case 2703: LAUNCH_KERNEL(2703); break;
	case 2704: LAUNCH_KERNEL(2704); break;
	case 2705: LAUNCH_KERNEL(2705); break;
	case 2706: LAUNCH_KERNEL(2706); break;
	case 2707: LAUNCH_KERNEL(2707); break;
	case 2708: LAUNCH_KERNEL(2708); break;
	case 2709: LAUNCH_KERNEL(2709); break;
	case 2710: LAUNCH_KERNEL(2710); break;
	case 2711: LAUNCH_KERNEL(2711); break;
	case 2712: LAUNCH_KERNEL(2712); break;
	case 2713: LAUNCH_KERNEL(2713); break;
	case 2714: LAUNCH_KERNEL(2714); break;
	case 2715: LAUNCH_KERNEL(2715); break;
	case 2716: LAUNCH_KERNEL(2716); break;
	case 2717: LAUNCH_KERNEL(2717); break;
	case 2718: LAUNCH_KERNEL(2718); break;
	case 2719: LAUNCH_KERNEL(2719); break;
	case 2720: LAUNCH_KERNEL(2720); break;
	case 2721: LAUNCH_KERNEL(2721); break;
	case 2722: LAUNCH_KERNEL(2722); break;
	case 2723: LAUNCH_KERNEL(2723); break;
	case 2724: LAUNCH_KERNEL(2724); break;
	case 2725: LAUNCH_KERNEL(2725); break;
	case 2726: LAUNCH_KERNEL(2726); break;
	case 2727: LAUNCH_KERNEL(2727); break;
	case 2728: LAUNCH_KERNEL(2728); break;
	case 2729: LAUNCH_KERNEL(2729); break;
	case 2730: LAUNCH_KERNEL(2730); break;
	case 2731: LAUNCH_KERNEL(2731); break;
	case 2732: LAUNCH_KERNEL(2732); break;
	case 2733: LAUNCH_KERNEL(2733); break;
	case 2734: LAUNCH_KERNEL(2734); break;
	case 2735: LAUNCH_KERNEL(2735); break;
	case 2736: LAUNCH_KERNEL(2736); break;
	case 2737: LAUNCH_KERNEL(2737); break;
	case 2738: LAUNCH_KERNEL(2738); break;
	case 2739: LAUNCH_KERNEL(2739); break;
	case 2740: LAUNCH_KERNEL(2740); break;
	case 2741: LAUNCH_KERNEL(2741); break;
	case 2742: LAUNCH_KERNEL(2742); break;
	case 2743: LAUNCH_KERNEL(2743); break;
	case 2744: LAUNCH_KERNEL(2744); break;
	case 2745: LAUNCH_KERNEL(2745); break;
	case 2746: LAUNCH_KERNEL(2746); break;
	case 2747: LAUNCH_KERNEL(2747); break;
	case 2748: LAUNCH_KERNEL(2748); break;
	case 2749: LAUNCH_KERNEL(2749); break;
	case 2750: LAUNCH_KERNEL(2750); break;
	case 2751: LAUNCH_KERNEL(2751); break;
	case 2752: LAUNCH_KERNEL(2752); break;
	case 2753: LAUNCH_KERNEL(2753); break;
	case 2754: LAUNCH_KERNEL(2754); break;
	case 2755: LAUNCH_KERNEL(2755); break;
	case 2756: LAUNCH_KERNEL(2756); break;
	case 2757: LAUNCH_KERNEL(2757); break;
	case 2758: LAUNCH_KERNEL(2758); break;
	case 2759: LAUNCH_KERNEL(2759); break;
	case 2760: LAUNCH_KERNEL(2760); break;
	case 2761: LAUNCH_KERNEL(2761); break;
	case 2762: LAUNCH_KERNEL(2762); break;
	case 2763: LAUNCH_KERNEL(2763); break;
	case 2764: LAUNCH_KERNEL(2764); break;
	case 2765: LAUNCH_KERNEL(2765); break;
	case 2766: LAUNCH_KERNEL(2766); break;
	case 2767: LAUNCH_KERNEL(2767); break;
	case 2768: LAUNCH_KERNEL(2768); break;
	case 2769: LAUNCH_KERNEL(2769); break;
	case 2770: LAUNCH_KERNEL(2770); break;
	case 2771: LAUNCH_KERNEL(2771); break;
	case 2772: LAUNCH_KERNEL(2772); break;
	case 2773: LAUNCH_KERNEL(2773); break;
	case 2774: LAUNCH_KERNEL(2774); break;
	case 2775: LAUNCH_KERNEL(2775); break;
	case 2776: LAUNCH_KERNEL(2776); break;
	case 2777: LAUNCH_KERNEL(2777); break;
	case 2778: LAUNCH_KERNEL(2778); break;
	case 2779: LAUNCH_KERNEL(2779); break;
	case 2780: LAUNCH_KERNEL(2780); break;
	case 2781: LAUNCH_KERNEL(2781); break;
	case 2782: LAUNCH_KERNEL(2782); break;
	case 2783: LAUNCH_KERNEL(2783); break;
	case 2784: LAUNCH_KERNEL(2784); break;
	case 2785: LAUNCH_KERNEL(2785); break;
	case 2786: LAUNCH_KERNEL(2786); break;
	case 2787: LAUNCH_KERNEL(2787); break;
	case 2788: LAUNCH_KERNEL(2788); break;
	case 2789: LAUNCH_KERNEL(2789); break;
	case 2790: LAUNCH_KERNEL(2790); break;
	case 2791: LAUNCH_KERNEL(2791); break;
	case 2792: LAUNCH_KERNEL(2792); break;
	case 2793: LAUNCH_KERNEL(2793); break;
	case 2794: LAUNCH_KERNEL(2794); break;
	case 2795: LAUNCH_KERNEL(2795); break;
	case 2796: LAUNCH_KERNEL(2796); break;
	case 2797: LAUNCH_KERNEL(2797); break;
	case 2798: LAUNCH_KERNEL(2798); break;
	case 2799: LAUNCH_KERNEL(2799); break;
	case 2800: LAUNCH_KERNEL(2800); break;
	case 2801: LAUNCH_KERNEL(2801); break;
	case 2802: LAUNCH_KERNEL(2802); break;
	case 2803: LAUNCH_KERNEL(2803); break;
	case 2804: LAUNCH_KERNEL(2804); break;
	case 2805: LAUNCH_KERNEL(2805); break;
	case 2806: LAUNCH_KERNEL(2806); break;
	case 2807: LAUNCH_KERNEL(2807); break;
	case 2808: LAUNCH_KERNEL(2808); break;
	case 2809: LAUNCH_KERNEL(2809); break;
	case 2810: LAUNCH_KERNEL(2810); break;
	case 2811: LAUNCH_KERNEL(2811); break;
	case 2812: LAUNCH_KERNEL(2812); break;
	case 2813: LAUNCH_KERNEL(2813); break;
	case 2814: LAUNCH_KERNEL(2814); break;
	case 2815: LAUNCH_KERNEL(2815); break;
	case 2816: LAUNCH_KERNEL(2816); break;
	case 2817: LAUNCH_KERNEL(2817); break;
	case 2818: LAUNCH_KERNEL(2818); break;
	case 2819: LAUNCH_KERNEL(2819); break;
	case 2820: LAUNCH_KERNEL(2820); break;
	case 2821: LAUNCH_KERNEL(2821); break;
	case 2822: LAUNCH_KERNEL(2822); break;
	case 2823: LAUNCH_KERNEL(2823); break;
	case 2824: LAUNCH_KERNEL(2824); break;
	case 2825: LAUNCH_KERNEL(2825); break;
	case 2826: LAUNCH_KERNEL(2826); break;
	case 2827: LAUNCH_KERNEL(2827); break;
	case 2828: LAUNCH_KERNEL(2828); break;
	case 2829: LAUNCH_KERNEL(2829); break;
	case 2830: LAUNCH_KERNEL(2830); break;
	case 2831: LAUNCH_KERNEL(2831); break;
	case 2832: LAUNCH_KERNEL(2832); break;
	case 2833: LAUNCH_KERNEL(2833); break;
	case 2834: LAUNCH_KERNEL(2834); break;
	case 2835: LAUNCH_KERNEL(2835); break;
	case 2836: LAUNCH_KERNEL(2836); break;
	case 2837: LAUNCH_KERNEL(2837); break;
	case 2838: LAUNCH_KERNEL(2838); break;
	case 2839: LAUNCH_KERNEL(2839); break;
	case 2840: LAUNCH_KERNEL(2840); break;
	case 2841: LAUNCH_KERNEL(2841); break;
	case 2842: LAUNCH_KERNEL(2842); break;
	case 2843: LAUNCH_KERNEL(2843); break;
	case 2844: LAUNCH_KERNEL(2844); break;
	case 2845: LAUNCH_KERNEL(2845); break;
	case 2846: LAUNCH_KERNEL(2846); break;
	case 2847: LAUNCH_KERNEL(2847); break;
	case 2848: LAUNCH_KERNEL(2848); break;
	case 2849: LAUNCH_KERNEL(2849); break;
	case 2850: LAUNCH_KERNEL(2850); break;
	case 2851: LAUNCH_KERNEL(2851); break;
	case 2852: LAUNCH_KERNEL(2852); break;
	case 2853: LAUNCH_KERNEL(2853); break;
	case 2854: LAUNCH_KERNEL(2854); break;
	case 2855: LAUNCH_KERNEL(2855); break;
	case 2856: LAUNCH_KERNEL(2856); break;
	case 2857: LAUNCH_KERNEL(2857); break;
	case 2858: LAUNCH_KERNEL(2858); break;
	case 2859: LAUNCH_KERNEL(2859); break;
	case 2860: LAUNCH_KERNEL(2860); break;
	case 2861: LAUNCH_KERNEL(2861); break;
	case 2862: LAUNCH_KERNEL(2862); break;
	case 2863: LAUNCH_KERNEL(2863); break;
	case 2864: LAUNCH_KERNEL(2864); break;
	case 2865: LAUNCH_KERNEL(2865); break;
	case 2866: LAUNCH_KERNEL(2866); break;
	case 2867: LAUNCH_KERNEL(2867); break;
	case 2868: LAUNCH_KERNEL(2868); break;
	case 2869: LAUNCH_KERNEL(2869); break;
	case 2870: LAUNCH_KERNEL(2870); break;
	case 2871: LAUNCH_KERNEL(2871); break;
	case 2872: LAUNCH_KERNEL(2872); break;
	case 2873: LAUNCH_KERNEL(2873); break;
	case 2874: LAUNCH_KERNEL(2874); break;
	case 2875: LAUNCH_KERNEL(2875); break;
	case 2876: LAUNCH_KERNEL(2876); break;
	case 2877: LAUNCH_KERNEL(2877); break;
	case 2878: LAUNCH_KERNEL(2878); break;
	case 2879: LAUNCH_KERNEL(2879); break;
	case 2880: LAUNCH_KERNEL(2880); break;
	case 2881: LAUNCH_KERNEL(2881); break;
	case 2882: LAUNCH_KERNEL(2882); break;
	case 2883: LAUNCH_KERNEL(2883); break;
	case 2884: LAUNCH_KERNEL(2884); break;
	case 2885: LAUNCH_KERNEL(2885); break;
	case 2886: LAUNCH_KERNEL(2886); break;
	case 2887: LAUNCH_KERNEL(2887); break;
	case 2888: LAUNCH_KERNEL(2888); break;
	case 2889: LAUNCH_KERNEL(2889); break;
	case 2890: LAUNCH_KERNEL(2890); break;
	case 2891: LAUNCH_KERNEL(2891); break;
	case 2892: LAUNCH_KERNEL(2892); break;
	case 2893: LAUNCH_KERNEL(2893); break;
	case 2894: LAUNCH_KERNEL(2894); break;
	case 2895: LAUNCH_KERNEL(2895); break;
	case 2896: LAUNCH_KERNEL(2896); break;
	case 2897: LAUNCH_KERNEL(2897); break;
	case 2898: LAUNCH_KERNEL(2898); break;
	case 2899: LAUNCH_KERNEL(2899); break;
	case 2900: LAUNCH_KERNEL(2900); break;
	case 2901: LAUNCH_KERNEL(2901); break;
	case 2902: LAUNCH_KERNEL(2902); break;
	case 2903: LAUNCH_KERNEL(2903); break;
	case 2904: LAUNCH_KERNEL(2904); break;
	case 2905: LAUNCH_KERNEL(2905); break;
	case 2906: LAUNCH_KERNEL(2906); break;
	case 2907: LAUNCH_KERNEL(2907); break;
	case 2908: LAUNCH_KERNEL(2908); break;
	case 2909: LAUNCH_KERNEL(2909); break;
	case 2910: LAUNCH_KERNEL(2910); break;
	case 2911: LAUNCH_KERNEL(2911); break;
	case 2912: LAUNCH_KERNEL(2912); break;
	case 2913: LAUNCH_KERNEL(2913); break;
	case 2914: LAUNCH_KERNEL(2914); break;
	case 2915: LAUNCH_KERNEL(2915); break;
	case 2916: LAUNCH_KERNEL(2916); break;
	case 2917: LAUNCH_KERNEL(2917); break;
	case 2918: LAUNCH_KERNEL(2918); break;
	case 2919: LAUNCH_KERNEL(2919); break;
	case 2920: LAUNCH_KERNEL(2920); break;
	case 2921: LAUNCH_KERNEL(2921); break;
	case 2922: LAUNCH_KERNEL(2922); break;
	case 2923: LAUNCH_KERNEL(2923); break;
	case 2924: LAUNCH_KERNEL(2924); break;
	case 2925: LAUNCH_KERNEL(2925); break;
	case 2926: LAUNCH_KERNEL(2926); break;
	case 2927: LAUNCH_KERNEL(2927); break;
	case 2928: LAUNCH_KERNEL(2928); break;
	case 2929: LAUNCH_KERNEL(2929); break;
	case 2930: LAUNCH_KERNEL(2930); break;
	case 2931: LAUNCH_KERNEL(2931); break;
	case 2932: LAUNCH_KERNEL(2932); break;
	case 2933: LAUNCH_KERNEL(2933); break;
	case 2934: LAUNCH_KERNEL(2934); break;
	case 2935: LAUNCH_KERNEL(2935); break;
	case 2936: LAUNCH_KERNEL(2936); break;
	case 2937: LAUNCH_KERNEL(2937); break;
	case 2938: LAUNCH_KERNEL(2938); break;
	case 2939: LAUNCH_KERNEL(2939); break;
	case 2940: LAUNCH_KERNEL(2940); break;
	case 2941: LAUNCH_KERNEL(2941); break;
	case 2942: LAUNCH_KERNEL(2942); break;
	case 2943: LAUNCH_KERNEL(2943); break;
	case 2944: LAUNCH_KERNEL(2944); break;
	case 2945: LAUNCH_KERNEL(2945); break;
	case 2946: LAUNCH_KERNEL(2946); break;
	case 2947: LAUNCH_KERNEL(2947); break;
	case 2948: LAUNCH_KERNEL(2948); break;
	case 2949: LAUNCH_KERNEL(2949); break;
	case 2950: LAUNCH_KERNEL(2950); break;
	case 2951: LAUNCH_KERNEL(2951); break;
	case 2952: LAUNCH_KERNEL(2952); break;
	case 2953: LAUNCH_KERNEL(2953); break;
	case 2954: LAUNCH_KERNEL(2954); break;
	case 2955: LAUNCH_KERNEL(2955); break;
	case 2956: LAUNCH_KERNEL(2956); break;
	case 2957: LAUNCH_KERNEL(2957); break;
	case 2958: LAUNCH_KERNEL(2958); break;
	case 2959: LAUNCH_KERNEL(2959); break;
	case 2960: LAUNCH_KERNEL(2960); break;
	case 2961: LAUNCH_KERNEL(2961); break;
	case 2962: LAUNCH_KERNEL(2962); break;
	case 2963: LAUNCH_KERNEL(2963); break;
	case 2964: LAUNCH_KERNEL(2964); break;
	case 2965: LAUNCH_KERNEL(2965); break;
	case 2966: LAUNCH_KERNEL(2966); break;
	case 2967: LAUNCH_KERNEL(2967); break;
	case 2968: LAUNCH_KERNEL(2968); break;
	case 2969: LAUNCH_KERNEL(2969); break;
	case 2970: LAUNCH_KERNEL(2970); break;
	case 2971: LAUNCH_KERNEL(2971); break;
	case 2972: LAUNCH_KERNEL(2972); break;
	case 2973: LAUNCH_KERNEL(2973); break;
	case 2974: LAUNCH_KERNEL(2974); break;
	case 2975: LAUNCH_KERNEL(2975); break;
	case 2976: LAUNCH_KERNEL(2976); break;
	case 2977: LAUNCH_KERNEL(2977); break;
	case 2978: LAUNCH_KERNEL(2978); break;
	case 2979: LAUNCH_KERNEL(2979); break;
	case 2980: LAUNCH_KERNEL(2980); break;
	case 2981: LAUNCH_KERNEL(2981); break;
	case 2982: LAUNCH_KERNEL(2982); break;
	case 2983: LAUNCH_KERNEL(2983); break;
	case 2984: LAUNCH_KERNEL(2984); break;
	case 2985: LAUNCH_KERNEL(2985); break;
	case 2986: LAUNCH_KERNEL(2986); break;
	case 2987: LAUNCH_KERNEL(2987); break;
	case 2988: LAUNCH_KERNEL(2988); break;
	case 2989: LAUNCH_KERNEL(2989); break;
	case 2990: LAUNCH_KERNEL(2990); break;
	case 2991: LAUNCH_KERNEL(2991); break;
	case 2992: LAUNCH_KERNEL(2992); break;
	case 2993: LAUNCH_KERNEL(2993); break;
	case 2994: LAUNCH_KERNEL(2994); break;
	case 2995: LAUNCH_KERNEL(2995); break;
	case 2996: LAUNCH_KERNEL(2996); break;
	case 2997: LAUNCH_KERNEL(2997); break;
	case 2998: LAUNCH_KERNEL(2998); break;
	case 2999: LAUNCH_KERNEL(2999); break;
	case 3000: LAUNCH_KERNEL(3000); break;
	case 3001: LAUNCH_KERNEL(3001); break;
	case 3002: LAUNCH_KERNEL(3002); break;
	case 3003: LAUNCH_KERNEL(3003); break;
	case 3004: LAUNCH_KERNEL(3004); break;
	case 3005: LAUNCH_KERNEL(3005); break;
	case 3006: LAUNCH_KERNEL(3006); break;
	case 3007: LAUNCH_KERNEL(3007); break;
	case 3008: LAUNCH_KERNEL(3008); break;
	case 3009: LAUNCH_KERNEL(3009); break;
	case 3010: LAUNCH_KERNEL(3010); break;
	case 3011: LAUNCH_KERNEL(3011); break;
	case 3012: LAUNCH_KERNEL(3012); break;
	case 3013: LAUNCH_KERNEL(3013); break;
	case 3014: LAUNCH_KERNEL(3014); break;
	case 3015: LAUNCH_KERNEL(3015); break;
	case 3016: LAUNCH_KERNEL(3016); break;
	case 3017: LAUNCH_KERNEL(3017); break;
	case 3018: LAUNCH_KERNEL(3018); break;
	case 3019: LAUNCH_KERNEL(3019); break;
	case 3020: LAUNCH_KERNEL(3020); break;
	case 3021: LAUNCH_KERNEL(3021); break;
	case 3022: LAUNCH_KERNEL(3022); break;
	case 3023: LAUNCH_KERNEL(3023); break;
	case 3024: LAUNCH_KERNEL(3024); break;
	case 3025: LAUNCH_KERNEL(3025); break;
	case 3026: LAUNCH_KERNEL(3026); break;
	case 3027: LAUNCH_KERNEL(3027); break;
	case 3028: LAUNCH_KERNEL(3028); break;
	case 3029: LAUNCH_KERNEL(3029); break;
	case 3030: LAUNCH_KERNEL(3030); break;
	case 3031: LAUNCH_KERNEL(3031); break;
	case 3032: LAUNCH_KERNEL(3032); break;
	case 3033: LAUNCH_KERNEL(3033); break;
	case 3034: LAUNCH_KERNEL(3034); break;
	case 3035: LAUNCH_KERNEL(3035); break;
	case 3036: LAUNCH_KERNEL(3036); break;
	case 3037: LAUNCH_KERNEL(3037); break;
	case 3038: LAUNCH_KERNEL(3038); break;
	case 3039: LAUNCH_KERNEL(3039); break;
	case 3040: LAUNCH_KERNEL(3040); break;
	case 3041: LAUNCH_KERNEL(3041); break;
	case 3042: LAUNCH_KERNEL(3042); break;
	case 3043: LAUNCH_KERNEL(3043); break;
	case 3044: LAUNCH_KERNEL(3044); break;
	case 3045: LAUNCH_KERNEL(3045); break;
	case 3046: LAUNCH_KERNEL(3046); break;
	case 3047: LAUNCH_KERNEL(3047); break;
	case 3048: LAUNCH_KERNEL(3048); break;
	case 3049: LAUNCH_KERNEL(3049); break;
	case 3050: LAUNCH_KERNEL(3050); break;
	case 3051: LAUNCH_KERNEL(3051); break;
	case 3052: LAUNCH_KERNEL(3052); break;
	case 3053: LAUNCH_KERNEL(3053); break;
	case 3054: LAUNCH_KERNEL(3054); break;
	case 3055: LAUNCH_KERNEL(3055); break;
	case 3056: LAUNCH_KERNEL(3056); break;
	case 3057: LAUNCH_KERNEL(3057); break;
	case 3058: LAUNCH_KERNEL(3058); break;
	case 3059: LAUNCH_KERNEL(3059); break;
	case 3060: LAUNCH_KERNEL(3060); break;
	case 3061: LAUNCH_KERNEL(3061); break;
	case 3062: LAUNCH_KERNEL(3062); break;
	case 3063: LAUNCH_KERNEL(3063); break;
	case 3064: LAUNCH_KERNEL(3064); break;
	case 3065: LAUNCH_KERNEL(3065); break;
	case 3066: LAUNCH_KERNEL(3066); break;
	case 3067: LAUNCH_KERNEL(3067); break;
	case 3068: LAUNCH_KERNEL(3068); break;
	case 3069: LAUNCH_KERNEL(3069); break;
	case 3070: LAUNCH_KERNEL(3070); break;
	case 3071: LAUNCH_KERNEL(3071); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
