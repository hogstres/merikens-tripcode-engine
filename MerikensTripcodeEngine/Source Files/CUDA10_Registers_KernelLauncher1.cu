// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 512
#include "CUDA10_Registers_Kernel.h"
#define SALT 513
#include "CUDA10_Registers_Kernel.h"
#define SALT 514
#include "CUDA10_Registers_Kernel.h"
#define SALT 515
#include "CUDA10_Registers_Kernel.h"
#define SALT 516
#include "CUDA10_Registers_Kernel.h"
#define SALT 517
#include "CUDA10_Registers_Kernel.h"
#define SALT 518
#include "CUDA10_Registers_Kernel.h"
#define SALT 519
#include "CUDA10_Registers_Kernel.h"
#define SALT 520
#include "CUDA10_Registers_Kernel.h"
#define SALT 521
#include "CUDA10_Registers_Kernel.h"
#define SALT 522
#include "CUDA10_Registers_Kernel.h"
#define SALT 523
#include "CUDA10_Registers_Kernel.h"
#define SALT 524
#include "CUDA10_Registers_Kernel.h"
#define SALT 525
#include "CUDA10_Registers_Kernel.h"
#define SALT 526
#include "CUDA10_Registers_Kernel.h"
#define SALT 527
#include "CUDA10_Registers_Kernel.h"
#define SALT 528
#include "CUDA10_Registers_Kernel.h"
#define SALT 529
#include "CUDA10_Registers_Kernel.h"
#define SALT 530
#include "CUDA10_Registers_Kernel.h"
#define SALT 531
#include "CUDA10_Registers_Kernel.h"
#define SALT 532
#include "CUDA10_Registers_Kernel.h"
#define SALT 533
#include "CUDA10_Registers_Kernel.h"
#define SALT 534
#include "CUDA10_Registers_Kernel.h"
#define SALT 535
#include "CUDA10_Registers_Kernel.h"
#define SALT 536
#include "CUDA10_Registers_Kernel.h"
#define SALT 537
#include "CUDA10_Registers_Kernel.h"
#define SALT 538
#include "CUDA10_Registers_Kernel.h"
#define SALT 539
#include "CUDA10_Registers_Kernel.h"
#define SALT 540
#include "CUDA10_Registers_Kernel.h"
#define SALT 541
#include "CUDA10_Registers_Kernel.h"
#define SALT 542
#include "CUDA10_Registers_Kernel.h"
#define SALT 543
#include "CUDA10_Registers_Kernel.h"
#define SALT 544
#include "CUDA10_Registers_Kernel.h"
#define SALT 545
#include "CUDA10_Registers_Kernel.h"
#define SALT 546
#include "CUDA10_Registers_Kernel.h"
#define SALT 547
#include "CUDA10_Registers_Kernel.h"
#define SALT 548
#include "CUDA10_Registers_Kernel.h"
#define SALT 549
#include "CUDA10_Registers_Kernel.h"
#define SALT 550
#include "CUDA10_Registers_Kernel.h"
#define SALT 551
#include "CUDA10_Registers_Kernel.h"
#define SALT 552
#include "CUDA10_Registers_Kernel.h"
#define SALT 553
#include "CUDA10_Registers_Kernel.h"
#define SALT 554
#include "CUDA10_Registers_Kernel.h"
#define SALT 555
#include "CUDA10_Registers_Kernel.h"
#define SALT 556
#include "CUDA10_Registers_Kernel.h"
#define SALT 557
#include "CUDA10_Registers_Kernel.h"
#define SALT 558
#include "CUDA10_Registers_Kernel.h"
#define SALT 559
#include "CUDA10_Registers_Kernel.h"
#define SALT 560
#include "CUDA10_Registers_Kernel.h"
#define SALT 561
#include "CUDA10_Registers_Kernel.h"
#define SALT 562
#include "CUDA10_Registers_Kernel.h"
#define SALT 563
#include "CUDA10_Registers_Kernel.h"
#define SALT 564
#include "CUDA10_Registers_Kernel.h"
#define SALT 565
#include "CUDA10_Registers_Kernel.h"
#define SALT 566
#include "CUDA10_Registers_Kernel.h"
#define SALT 567
#include "CUDA10_Registers_Kernel.h"
#define SALT 568
#include "CUDA10_Registers_Kernel.h"
#define SALT 569
#include "CUDA10_Registers_Kernel.h"
#define SALT 570
#include "CUDA10_Registers_Kernel.h"
#define SALT 571
#include "CUDA10_Registers_Kernel.h"
#define SALT 572
#include "CUDA10_Registers_Kernel.h"
#define SALT 573
#include "CUDA10_Registers_Kernel.h"
#define SALT 574
#include "CUDA10_Registers_Kernel.h"
#define SALT 575
#include "CUDA10_Registers_Kernel.h"
#define SALT 576
#include "CUDA10_Registers_Kernel.h"
#define SALT 577
#include "CUDA10_Registers_Kernel.h"
#define SALT 578
#include "CUDA10_Registers_Kernel.h"
#define SALT 579
#include "CUDA10_Registers_Kernel.h"
#define SALT 580
#include "CUDA10_Registers_Kernel.h"
#define SALT 581
#include "CUDA10_Registers_Kernel.h"
#define SALT 582
#include "CUDA10_Registers_Kernel.h"
#define SALT 583
#include "CUDA10_Registers_Kernel.h"
#define SALT 584
#include "CUDA10_Registers_Kernel.h"
#define SALT 585
#include "CUDA10_Registers_Kernel.h"
#define SALT 586
#include "CUDA10_Registers_Kernel.h"
#define SALT 587
#include "CUDA10_Registers_Kernel.h"
#define SALT 588
#include "CUDA10_Registers_Kernel.h"
#define SALT 589
#include "CUDA10_Registers_Kernel.h"
#define SALT 590
#include "CUDA10_Registers_Kernel.h"
#define SALT 591
#include "CUDA10_Registers_Kernel.h"
#define SALT 592
#include "CUDA10_Registers_Kernel.h"
#define SALT 593
#include "CUDA10_Registers_Kernel.h"
#define SALT 594
#include "CUDA10_Registers_Kernel.h"
#define SALT 595
#include "CUDA10_Registers_Kernel.h"
#define SALT 596
#include "CUDA10_Registers_Kernel.h"
#define SALT 597
#include "CUDA10_Registers_Kernel.h"
#define SALT 598
#include "CUDA10_Registers_Kernel.h"
#define SALT 599
#include "CUDA10_Registers_Kernel.h"
#define SALT 600
#include "CUDA10_Registers_Kernel.h"
#define SALT 601
#include "CUDA10_Registers_Kernel.h"
#define SALT 602
#include "CUDA10_Registers_Kernel.h"
#define SALT 603
#include "CUDA10_Registers_Kernel.h"
#define SALT 604
#include "CUDA10_Registers_Kernel.h"
#define SALT 605
#include "CUDA10_Registers_Kernel.h"
#define SALT 606
#include "CUDA10_Registers_Kernel.h"
#define SALT 607
#include "CUDA10_Registers_Kernel.h"
#define SALT 608
#include "CUDA10_Registers_Kernel.h"
#define SALT 609
#include "CUDA10_Registers_Kernel.h"
#define SALT 610
#include "CUDA10_Registers_Kernel.h"
#define SALT 611
#include "CUDA10_Registers_Kernel.h"
#define SALT 612
#include "CUDA10_Registers_Kernel.h"
#define SALT 613
#include "CUDA10_Registers_Kernel.h"
#define SALT 614
#include "CUDA10_Registers_Kernel.h"
#define SALT 615
#include "CUDA10_Registers_Kernel.h"
#define SALT 616
#include "CUDA10_Registers_Kernel.h"
#define SALT 617
#include "CUDA10_Registers_Kernel.h"
#define SALT 618
#include "CUDA10_Registers_Kernel.h"
#define SALT 619
#include "CUDA10_Registers_Kernel.h"
#define SALT 620
#include "CUDA10_Registers_Kernel.h"
#define SALT 621
#include "CUDA10_Registers_Kernel.h"
#define SALT 622
#include "CUDA10_Registers_Kernel.h"
#define SALT 623
#include "CUDA10_Registers_Kernel.h"
#define SALT 624
#include "CUDA10_Registers_Kernel.h"
#define SALT 625
#include "CUDA10_Registers_Kernel.h"
#define SALT 626
#include "CUDA10_Registers_Kernel.h"
#define SALT 627
#include "CUDA10_Registers_Kernel.h"
#define SALT 628
#include "CUDA10_Registers_Kernel.h"
#define SALT 629
#include "CUDA10_Registers_Kernel.h"
#define SALT 630
#include "CUDA10_Registers_Kernel.h"
#define SALT 631
#include "CUDA10_Registers_Kernel.h"
#define SALT 632
#include "CUDA10_Registers_Kernel.h"
#define SALT 633
#include "CUDA10_Registers_Kernel.h"
#define SALT 634
#include "CUDA10_Registers_Kernel.h"
#define SALT 635
#include "CUDA10_Registers_Kernel.h"
#define SALT 636
#include "CUDA10_Registers_Kernel.h"
#define SALT 637
#include "CUDA10_Registers_Kernel.h"
#define SALT 638
#include "CUDA10_Registers_Kernel.h"
#define SALT 639
#include "CUDA10_Registers_Kernel.h"
#define SALT 640
#include "CUDA10_Registers_Kernel.h"
#define SALT 641
#include "CUDA10_Registers_Kernel.h"
#define SALT 642
#include "CUDA10_Registers_Kernel.h"
#define SALT 643
#include "CUDA10_Registers_Kernel.h"
#define SALT 644
#include "CUDA10_Registers_Kernel.h"
#define SALT 645
#include "CUDA10_Registers_Kernel.h"
#define SALT 646
#include "CUDA10_Registers_Kernel.h"
#define SALT 647
#include "CUDA10_Registers_Kernel.h"
#define SALT 648
#include "CUDA10_Registers_Kernel.h"
#define SALT 649
#include "CUDA10_Registers_Kernel.h"
#define SALT 650
#include "CUDA10_Registers_Kernel.h"
#define SALT 651
#include "CUDA10_Registers_Kernel.h"
#define SALT 652
#include "CUDA10_Registers_Kernel.h"
#define SALT 653
#include "CUDA10_Registers_Kernel.h"
#define SALT 654
#include "CUDA10_Registers_Kernel.h"
#define SALT 655
#include "CUDA10_Registers_Kernel.h"
#define SALT 656
#include "CUDA10_Registers_Kernel.h"
#define SALT 657
#include "CUDA10_Registers_Kernel.h"
#define SALT 658
#include "CUDA10_Registers_Kernel.h"
#define SALT 659
#include "CUDA10_Registers_Kernel.h"
#define SALT 660
#include "CUDA10_Registers_Kernel.h"
#define SALT 661
#include "CUDA10_Registers_Kernel.h"
#define SALT 662
#include "CUDA10_Registers_Kernel.h"
#define SALT 663
#include "CUDA10_Registers_Kernel.h"
#define SALT 664
#include "CUDA10_Registers_Kernel.h"
#define SALT 665
#include "CUDA10_Registers_Kernel.h"
#define SALT 666
#include "CUDA10_Registers_Kernel.h"
#define SALT 667
#include "CUDA10_Registers_Kernel.h"
#define SALT 668
#include "CUDA10_Registers_Kernel.h"
#define SALT 669
#include "CUDA10_Registers_Kernel.h"
#define SALT 670
#include "CUDA10_Registers_Kernel.h"
#define SALT 671
#include "CUDA10_Registers_Kernel.h"
#define SALT 672
#include "CUDA10_Registers_Kernel.h"
#define SALT 673
#include "CUDA10_Registers_Kernel.h"
#define SALT 674
#include "CUDA10_Registers_Kernel.h"
#define SALT 675
#include "CUDA10_Registers_Kernel.h"
#define SALT 676
#include "CUDA10_Registers_Kernel.h"
#define SALT 677
#include "CUDA10_Registers_Kernel.h"
#define SALT 678
#include "CUDA10_Registers_Kernel.h"
#define SALT 679
#include "CUDA10_Registers_Kernel.h"
#define SALT 680
#include "CUDA10_Registers_Kernel.h"
#define SALT 681
#include "CUDA10_Registers_Kernel.h"
#define SALT 682
#include "CUDA10_Registers_Kernel.h"
#define SALT 683
#include "CUDA10_Registers_Kernel.h"
#define SALT 684
#include "CUDA10_Registers_Kernel.h"
#define SALT 685
#include "CUDA10_Registers_Kernel.h"
#define SALT 686
#include "CUDA10_Registers_Kernel.h"
#define SALT 687
#include "CUDA10_Registers_Kernel.h"
#define SALT 688
#include "CUDA10_Registers_Kernel.h"
#define SALT 689
#include "CUDA10_Registers_Kernel.h"
#define SALT 690
#include "CUDA10_Registers_Kernel.h"
#define SALT 691
#include "CUDA10_Registers_Kernel.h"
#define SALT 692
#include "CUDA10_Registers_Kernel.h"
#define SALT 693
#include "CUDA10_Registers_Kernel.h"
#define SALT 694
#include "CUDA10_Registers_Kernel.h"
#define SALT 695
#include "CUDA10_Registers_Kernel.h"
#define SALT 696
#include "CUDA10_Registers_Kernel.h"
#define SALT 697
#include "CUDA10_Registers_Kernel.h"
#define SALT 698
#include "CUDA10_Registers_Kernel.h"
#define SALT 699
#include "CUDA10_Registers_Kernel.h"
#define SALT 700
#include "CUDA10_Registers_Kernel.h"
#define SALT 701
#include "CUDA10_Registers_Kernel.h"
#define SALT 702
#include "CUDA10_Registers_Kernel.h"
#define SALT 703
#include "CUDA10_Registers_Kernel.h"
#define SALT 704
#include "CUDA10_Registers_Kernel.h"
#define SALT 705
#include "CUDA10_Registers_Kernel.h"
#define SALT 706
#include "CUDA10_Registers_Kernel.h"
#define SALT 707
#include "CUDA10_Registers_Kernel.h"
#define SALT 708
#include "CUDA10_Registers_Kernel.h"
#define SALT 709
#include "CUDA10_Registers_Kernel.h"
#define SALT 710
#include "CUDA10_Registers_Kernel.h"
#define SALT 711
#include "CUDA10_Registers_Kernel.h"
#define SALT 712
#include "CUDA10_Registers_Kernel.h"
#define SALT 713
#include "CUDA10_Registers_Kernel.h"
#define SALT 714
#include "CUDA10_Registers_Kernel.h"
#define SALT 715
#include "CUDA10_Registers_Kernel.h"
#define SALT 716
#include "CUDA10_Registers_Kernel.h"
#define SALT 717
#include "CUDA10_Registers_Kernel.h"
#define SALT 718
#include "CUDA10_Registers_Kernel.h"
#define SALT 719
#include "CUDA10_Registers_Kernel.h"
#define SALT 720
#include "CUDA10_Registers_Kernel.h"
#define SALT 721
#include "CUDA10_Registers_Kernel.h"
#define SALT 722
#include "CUDA10_Registers_Kernel.h"
#define SALT 723
#include "CUDA10_Registers_Kernel.h"
#define SALT 724
#include "CUDA10_Registers_Kernel.h"
#define SALT 725
#include "CUDA10_Registers_Kernel.h"
#define SALT 726
#include "CUDA10_Registers_Kernel.h"
#define SALT 727
#include "CUDA10_Registers_Kernel.h"
#define SALT 728
#include "CUDA10_Registers_Kernel.h"
#define SALT 729
#include "CUDA10_Registers_Kernel.h"
#define SALT 730
#include "CUDA10_Registers_Kernel.h"
#define SALT 731
#include "CUDA10_Registers_Kernel.h"
#define SALT 732
#include "CUDA10_Registers_Kernel.h"
#define SALT 733
#include "CUDA10_Registers_Kernel.h"
#define SALT 734
#include "CUDA10_Registers_Kernel.h"
#define SALT 735
#include "CUDA10_Registers_Kernel.h"
#define SALT 736
#include "CUDA10_Registers_Kernel.h"
#define SALT 737
#include "CUDA10_Registers_Kernel.h"
#define SALT 738
#include "CUDA10_Registers_Kernel.h"
#define SALT 739
#include "CUDA10_Registers_Kernel.h"
#define SALT 740
#include "CUDA10_Registers_Kernel.h"
#define SALT 741
#include "CUDA10_Registers_Kernel.h"
#define SALT 742
#include "CUDA10_Registers_Kernel.h"
#define SALT 743
#include "CUDA10_Registers_Kernel.h"
#define SALT 744
#include "CUDA10_Registers_Kernel.h"
#define SALT 745
#include "CUDA10_Registers_Kernel.h"
#define SALT 746
#include "CUDA10_Registers_Kernel.h"
#define SALT 747
#include "CUDA10_Registers_Kernel.h"
#define SALT 748
#include "CUDA10_Registers_Kernel.h"
#define SALT 749
#include "CUDA10_Registers_Kernel.h"
#define SALT 750
#include "CUDA10_Registers_Kernel.h"
#define SALT 751
#include "CUDA10_Registers_Kernel.h"
#define SALT 752
#include "CUDA10_Registers_Kernel.h"
#define SALT 753
#include "CUDA10_Registers_Kernel.h"
#define SALT 754
#include "CUDA10_Registers_Kernel.h"
#define SALT 755
#include "CUDA10_Registers_Kernel.h"
#define SALT 756
#include "CUDA10_Registers_Kernel.h"
#define SALT 757
#include "CUDA10_Registers_Kernel.h"
#define SALT 758
#include "CUDA10_Registers_Kernel.h"
#define SALT 759
#include "CUDA10_Registers_Kernel.h"
#define SALT 760
#include "CUDA10_Registers_Kernel.h"
#define SALT 761
#include "CUDA10_Registers_Kernel.h"
#define SALT 762
#include "CUDA10_Registers_Kernel.h"
#define SALT 763
#include "CUDA10_Registers_Kernel.h"
#define SALT 764
#include "CUDA10_Registers_Kernel.h"
#define SALT 765
#include "CUDA10_Registers_Kernel.h"
#define SALT 766
#include "CUDA10_Registers_Kernel.h"
#define SALT 767
#include "CUDA10_Registers_Kernel.h"
#define SALT 768
#include "CUDA10_Registers_Kernel.h"
#define SALT 769
#include "CUDA10_Registers_Kernel.h"
#define SALT 770
#include "CUDA10_Registers_Kernel.h"
#define SALT 771
#include "CUDA10_Registers_Kernel.h"
#define SALT 772
#include "CUDA10_Registers_Kernel.h"
#define SALT 773
#include "CUDA10_Registers_Kernel.h"
#define SALT 774
#include "CUDA10_Registers_Kernel.h"
#define SALT 775
#include "CUDA10_Registers_Kernel.h"
#define SALT 776
#include "CUDA10_Registers_Kernel.h"
#define SALT 777
#include "CUDA10_Registers_Kernel.h"
#define SALT 778
#include "CUDA10_Registers_Kernel.h"
#define SALT 779
#include "CUDA10_Registers_Kernel.h"
#define SALT 780
#include "CUDA10_Registers_Kernel.h"
#define SALT 781
#include "CUDA10_Registers_Kernel.h"
#define SALT 782
#include "CUDA10_Registers_Kernel.h"
#define SALT 783
#include "CUDA10_Registers_Kernel.h"
#define SALT 784
#include "CUDA10_Registers_Kernel.h"
#define SALT 785
#include "CUDA10_Registers_Kernel.h"
#define SALT 786
#include "CUDA10_Registers_Kernel.h"
#define SALT 787
#include "CUDA10_Registers_Kernel.h"
#define SALT 788
#include "CUDA10_Registers_Kernel.h"
#define SALT 789
#include "CUDA10_Registers_Kernel.h"
#define SALT 790
#include "CUDA10_Registers_Kernel.h"
#define SALT 791
#include "CUDA10_Registers_Kernel.h"
#define SALT 792
#include "CUDA10_Registers_Kernel.h"
#define SALT 793
#include "CUDA10_Registers_Kernel.h"
#define SALT 794
#include "CUDA10_Registers_Kernel.h"
#define SALT 795
#include "CUDA10_Registers_Kernel.h"
#define SALT 796
#include "CUDA10_Registers_Kernel.h"
#define SALT 797
#include "CUDA10_Registers_Kernel.h"
#define SALT 798
#include "CUDA10_Registers_Kernel.h"
#define SALT 799
#include "CUDA10_Registers_Kernel.h"
#define SALT 800
#include "CUDA10_Registers_Kernel.h"
#define SALT 801
#include "CUDA10_Registers_Kernel.h"
#define SALT 802
#include "CUDA10_Registers_Kernel.h"
#define SALT 803
#include "CUDA10_Registers_Kernel.h"
#define SALT 804
#include "CUDA10_Registers_Kernel.h"
#define SALT 805
#include "CUDA10_Registers_Kernel.h"
#define SALT 806
#include "CUDA10_Registers_Kernel.h"
#define SALT 807
#include "CUDA10_Registers_Kernel.h"
#define SALT 808
#include "CUDA10_Registers_Kernel.h"
#define SALT 809
#include "CUDA10_Registers_Kernel.h"
#define SALT 810
#include "CUDA10_Registers_Kernel.h"
#define SALT 811
#include "CUDA10_Registers_Kernel.h"
#define SALT 812
#include "CUDA10_Registers_Kernel.h"
#define SALT 813
#include "CUDA10_Registers_Kernel.h"
#define SALT 814
#include "CUDA10_Registers_Kernel.h"
#define SALT 815
#include "CUDA10_Registers_Kernel.h"
#define SALT 816
#include "CUDA10_Registers_Kernel.h"
#define SALT 817
#include "CUDA10_Registers_Kernel.h"
#define SALT 818
#include "CUDA10_Registers_Kernel.h"
#define SALT 819
#include "CUDA10_Registers_Kernel.h"
#define SALT 820
#include "CUDA10_Registers_Kernel.h"
#define SALT 821
#include "CUDA10_Registers_Kernel.h"
#define SALT 822
#include "CUDA10_Registers_Kernel.h"
#define SALT 823
#include "CUDA10_Registers_Kernel.h"
#define SALT 824
#include "CUDA10_Registers_Kernel.h"
#define SALT 825
#include "CUDA10_Registers_Kernel.h"
#define SALT 826
#include "CUDA10_Registers_Kernel.h"
#define SALT 827
#include "CUDA10_Registers_Kernel.h"
#define SALT 828
#include "CUDA10_Registers_Kernel.h"
#define SALT 829
#include "CUDA10_Registers_Kernel.h"
#define SALT 830
#include "CUDA10_Registers_Kernel.h"
#define SALT 831
#include "CUDA10_Registers_Kernel.h"
#define SALT 832
#include "CUDA10_Registers_Kernel.h"
#define SALT 833
#include "CUDA10_Registers_Kernel.h"
#define SALT 834
#include "CUDA10_Registers_Kernel.h"
#define SALT 835
#include "CUDA10_Registers_Kernel.h"
#define SALT 836
#include "CUDA10_Registers_Kernel.h"
#define SALT 837
#include "CUDA10_Registers_Kernel.h"
#define SALT 838
#include "CUDA10_Registers_Kernel.h"
#define SALT 839
#include "CUDA10_Registers_Kernel.h"
#define SALT 840
#include "CUDA10_Registers_Kernel.h"
#define SALT 841
#include "CUDA10_Registers_Kernel.h"
#define SALT 842
#include "CUDA10_Registers_Kernel.h"
#define SALT 843
#include "CUDA10_Registers_Kernel.h"
#define SALT 844
#include "CUDA10_Registers_Kernel.h"
#define SALT 845
#include "CUDA10_Registers_Kernel.h"
#define SALT 846
#include "CUDA10_Registers_Kernel.h"
#define SALT 847
#include "CUDA10_Registers_Kernel.h"
#define SALT 848
#include "CUDA10_Registers_Kernel.h"
#define SALT 849
#include "CUDA10_Registers_Kernel.h"
#define SALT 850
#include "CUDA10_Registers_Kernel.h"
#define SALT 851
#include "CUDA10_Registers_Kernel.h"
#define SALT 852
#include "CUDA10_Registers_Kernel.h"
#define SALT 853
#include "CUDA10_Registers_Kernel.h"
#define SALT 854
#include "CUDA10_Registers_Kernel.h"
#define SALT 855
#include "CUDA10_Registers_Kernel.h"
#define SALT 856
#include "CUDA10_Registers_Kernel.h"
#define SALT 857
#include "CUDA10_Registers_Kernel.h"
#define SALT 858
#include "CUDA10_Registers_Kernel.h"
#define SALT 859
#include "CUDA10_Registers_Kernel.h"
#define SALT 860
#include "CUDA10_Registers_Kernel.h"
#define SALT 861
#include "CUDA10_Registers_Kernel.h"
#define SALT 862
#include "CUDA10_Registers_Kernel.h"
#define SALT 863
#include "CUDA10_Registers_Kernel.h"
#define SALT 864
#include "CUDA10_Registers_Kernel.h"
#define SALT 865
#include "CUDA10_Registers_Kernel.h"
#define SALT 866
#include "CUDA10_Registers_Kernel.h"
#define SALT 867
#include "CUDA10_Registers_Kernel.h"
#define SALT 868
#include "CUDA10_Registers_Kernel.h"
#define SALT 869
#include "CUDA10_Registers_Kernel.h"
#define SALT 870
#include "CUDA10_Registers_Kernel.h"
#define SALT 871
#include "CUDA10_Registers_Kernel.h"
#define SALT 872
#include "CUDA10_Registers_Kernel.h"
#define SALT 873
#include "CUDA10_Registers_Kernel.h"
#define SALT 874
#include "CUDA10_Registers_Kernel.h"
#define SALT 875
#include "CUDA10_Registers_Kernel.h"
#define SALT 876
#include "CUDA10_Registers_Kernel.h"
#define SALT 877
#include "CUDA10_Registers_Kernel.h"
#define SALT 878
#include "CUDA10_Registers_Kernel.h"
#define SALT 879
#include "CUDA10_Registers_Kernel.h"
#define SALT 880
#include "CUDA10_Registers_Kernel.h"
#define SALT 881
#include "CUDA10_Registers_Kernel.h"
#define SALT 882
#include "CUDA10_Registers_Kernel.h"
#define SALT 883
#include "CUDA10_Registers_Kernel.h"
#define SALT 884
#include "CUDA10_Registers_Kernel.h"
#define SALT 885
#include "CUDA10_Registers_Kernel.h"
#define SALT 886
#include "CUDA10_Registers_Kernel.h"
#define SALT 887
#include "CUDA10_Registers_Kernel.h"
#define SALT 888
#include "CUDA10_Registers_Kernel.h"
#define SALT 889
#include "CUDA10_Registers_Kernel.h"
#define SALT 890
#include "CUDA10_Registers_Kernel.h"
#define SALT 891
#include "CUDA10_Registers_Kernel.h"
#define SALT 892
#include "CUDA10_Registers_Kernel.h"
#define SALT 893
#include "CUDA10_Registers_Kernel.h"
#define SALT 894
#include "CUDA10_Registers_Kernel.h"
#define SALT 895
#include "CUDA10_Registers_Kernel.h"
#define SALT 896
#include "CUDA10_Registers_Kernel.h"
#define SALT 897
#include "CUDA10_Registers_Kernel.h"
#define SALT 898
#include "CUDA10_Registers_Kernel.h"
#define SALT 899
#include "CUDA10_Registers_Kernel.h"
#define SALT 900
#include "CUDA10_Registers_Kernel.h"
#define SALT 901
#include "CUDA10_Registers_Kernel.h"
#define SALT 902
#include "CUDA10_Registers_Kernel.h"
#define SALT 903
#include "CUDA10_Registers_Kernel.h"
#define SALT 904
#include "CUDA10_Registers_Kernel.h"
#define SALT 905
#include "CUDA10_Registers_Kernel.h"
#define SALT 906
#include "CUDA10_Registers_Kernel.h"
#define SALT 907
#include "CUDA10_Registers_Kernel.h"
#define SALT 908
#include "CUDA10_Registers_Kernel.h"
#define SALT 909
#include "CUDA10_Registers_Kernel.h"
#define SALT 910
#include "CUDA10_Registers_Kernel.h"
#define SALT 911
#include "CUDA10_Registers_Kernel.h"
#define SALT 912
#include "CUDA10_Registers_Kernel.h"
#define SALT 913
#include "CUDA10_Registers_Kernel.h"
#define SALT 914
#include "CUDA10_Registers_Kernel.h"
#define SALT 915
#include "CUDA10_Registers_Kernel.h"
#define SALT 916
#include "CUDA10_Registers_Kernel.h"
#define SALT 917
#include "CUDA10_Registers_Kernel.h"
#define SALT 918
#include "CUDA10_Registers_Kernel.h"
#define SALT 919
#include "CUDA10_Registers_Kernel.h"
#define SALT 920
#include "CUDA10_Registers_Kernel.h"
#define SALT 921
#include "CUDA10_Registers_Kernel.h"
#define SALT 922
#include "CUDA10_Registers_Kernel.h"
#define SALT 923
#include "CUDA10_Registers_Kernel.h"
#define SALT 924
#include "CUDA10_Registers_Kernel.h"
#define SALT 925
#include "CUDA10_Registers_Kernel.h"
#define SALT 926
#include "CUDA10_Registers_Kernel.h"
#define SALT 927
#include "CUDA10_Registers_Kernel.h"
#define SALT 928
#include "CUDA10_Registers_Kernel.h"
#define SALT 929
#include "CUDA10_Registers_Kernel.h"
#define SALT 930
#include "CUDA10_Registers_Kernel.h"
#define SALT 931
#include "CUDA10_Registers_Kernel.h"
#define SALT 932
#include "CUDA10_Registers_Kernel.h"
#define SALT 933
#include "CUDA10_Registers_Kernel.h"
#define SALT 934
#include "CUDA10_Registers_Kernel.h"
#define SALT 935
#include "CUDA10_Registers_Kernel.h"
#define SALT 936
#include "CUDA10_Registers_Kernel.h"
#define SALT 937
#include "CUDA10_Registers_Kernel.h"
#define SALT 938
#include "CUDA10_Registers_Kernel.h"
#define SALT 939
#include "CUDA10_Registers_Kernel.h"
#define SALT 940
#include "CUDA10_Registers_Kernel.h"
#define SALT 941
#include "CUDA10_Registers_Kernel.h"
#define SALT 942
#include "CUDA10_Registers_Kernel.h"
#define SALT 943
#include "CUDA10_Registers_Kernel.h"
#define SALT 944
#include "CUDA10_Registers_Kernel.h"
#define SALT 945
#include "CUDA10_Registers_Kernel.h"
#define SALT 946
#include "CUDA10_Registers_Kernel.h"
#define SALT 947
#include "CUDA10_Registers_Kernel.h"
#define SALT 948
#include "CUDA10_Registers_Kernel.h"
#define SALT 949
#include "CUDA10_Registers_Kernel.h"
#define SALT 950
#include "CUDA10_Registers_Kernel.h"
#define SALT 951
#include "CUDA10_Registers_Kernel.h"
#define SALT 952
#include "CUDA10_Registers_Kernel.h"
#define SALT 953
#include "CUDA10_Registers_Kernel.h"
#define SALT 954
#include "CUDA10_Registers_Kernel.h"
#define SALT 955
#include "CUDA10_Registers_Kernel.h"
#define SALT 956
#include "CUDA10_Registers_Kernel.h"
#define SALT 957
#include "CUDA10_Registers_Kernel.h"
#define SALT 958
#include "CUDA10_Registers_Kernel.h"
#define SALT 959
#include "CUDA10_Registers_Kernel.h"
#define SALT 960
#include "CUDA10_Registers_Kernel.h"
#define SALT 961
#include "CUDA10_Registers_Kernel.h"
#define SALT 962
#include "CUDA10_Registers_Kernel.h"
#define SALT 963
#include "CUDA10_Registers_Kernel.h"
#define SALT 964
#include "CUDA10_Registers_Kernel.h"
#define SALT 965
#include "CUDA10_Registers_Kernel.h"
#define SALT 966
#include "CUDA10_Registers_Kernel.h"
#define SALT 967
#include "CUDA10_Registers_Kernel.h"
#define SALT 968
#include "CUDA10_Registers_Kernel.h"
#define SALT 969
#include "CUDA10_Registers_Kernel.h"
#define SALT 970
#include "CUDA10_Registers_Kernel.h"
#define SALT 971
#include "CUDA10_Registers_Kernel.h"
#define SALT 972
#include "CUDA10_Registers_Kernel.h"
#define SALT 973
#include "CUDA10_Registers_Kernel.h"
#define SALT 974
#include "CUDA10_Registers_Kernel.h"
#define SALT 975
#include "CUDA10_Registers_Kernel.h"
#define SALT 976
#include "CUDA10_Registers_Kernel.h"
#define SALT 977
#include "CUDA10_Registers_Kernel.h"
#define SALT 978
#include "CUDA10_Registers_Kernel.h"
#define SALT 979
#include "CUDA10_Registers_Kernel.h"
#define SALT 980
#include "CUDA10_Registers_Kernel.h"
#define SALT 981
#include "CUDA10_Registers_Kernel.h"
#define SALT 982
#include "CUDA10_Registers_Kernel.h"
#define SALT 983
#include "CUDA10_Registers_Kernel.h"
#define SALT 984
#include "CUDA10_Registers_Kernel.h"
#define SALT 985
#include "CUDA10_Registers_Kernel.h"
#define SALT 986
#include "CUDA10_Registers_Kernel.h"
#define SALT 987
#include "CUDA10_Registers_Kernel.h"
#define SALT 988
#include "CUDA10_Registers_Kernel.h"
#define SALT 989
#include "CUDA10_Registers_Kernel.h"
#define SALT 990
#include "CUDA10_Registers_Kernel.h"
#define SALT 991
#include "CUDA10_Registers_Kernel.h"
#define SALT 992
#include "CUDA10_Registers_Kernel.h"
#define SALT 993
#include "CUDA10_Registers_Kernel.h"
#define SALT 994
#include "CUDA10_Registers_Kernel.h"
#define SALT 995
#include "CUDA10_Registers_Kernel.h"
#define SALT 996
#include "CUDA10_Registers_Kernel.h"
#define SALT 997
#include "CUDA10_Registers_Kernel.h"
#define SALT 998
#include "CUDA10_Registers_Kernel.h"
#define SALT 999
#include "CUDA10_Registers_Kernel.h"
#define SALT 1000
#include "CUDA10_Registers_Kernel.h"
#define SALT 1001
#include "CUDA10_Registers_Kernel.h"
#define SALT 1002
#include "CUDA10_Registers_Kernel.h"
#define SALT 1003
#include "CUDA10_Registers_Kernel.h"
#define SALT 1004
#include "CUDA10_Registers_Kernel.h"
#define SALT 1005
#include "CUDA10_Registers_Kernel.h"
#define SALT 1006
#include "CUDA10_Registers_Kernel.h"
#define SALT 1007
#include "CUDA10_Registers_Kernel.h"
#define SALT 1008
#include "CUDA10_Registers_Kernel.h"
#define SALT 1009
#include "CUDA10_Registers_Kernel.h"
#define SALT 1010
#include "CUDA10_Registers_Kernel.h"
#define SALT 1011
#include "CUDA10_Registers_Kernel.h"
#define SALT 1012
#include "CUDA10_Registers_Kernel.h"
#define SALT 1013
#include "CUDA10_Registers_Kernel.h"
#define SALT 1014
#include "CUDA10_Registers_Kernel.h"
#define SALT 1015
#include "CUDA10_Registers_Kernel.h"
#define SALT 1016
#include "CUDA10_Registers_Kernel.h"
#define SALT 1017
#include "CUDA10_Registers_Kernel.h"
#define SALT 1018
#include "CUDA10_Registers_Kernel.h"
#define SALT 1019
#include "CUDA10_Registers_Kernel.h"
#define SALT 1020
#include "CUDA10_Registers_Kernel.h"
#define SALT 1021
#include "CUDA10_Registers_Kernel.h"
#define SALT 1022
#include "CUDA10_Registers_Kernel.h"
#define SALT 1023
#include "CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher1()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel1(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
		case 512: LAUNCH_KERNEL(512); break;
		case 513: LAUNCH_KERNEL(513); break;
		case 514: LAUNCH_KERNEL(514); break;
		case 515: LAUNCH_KERNEL(515); break;
		case 516: LAUNCH_KERNEL(516); break;
		case 517: LAUNCH_KERNEL(517); break;
		case 518: LAUNCH_KERNEL(518); break;
		case 519: LAUNCH_KERNEL(519); break;
		case 520: LAUNCH_KERNEL(520); break;
		case 521: LAUNCH_KERNEL(521); break;
		case 522: LAUNCH_KERNEL(522); break;
		case 523: LAUNCH_KERNEL(523); break;
		case 524: LAUNCH_KERNEL(524); break;
		case 525: LAUNCH_KERNEL(525); break;
		case 526: LAUNCH_KERNEL(526); break;
		case 527: LAUNCH_KERNEL(527); break;
		case 528: LAUNCH_KERNEL(528); break;
		case 529: LAUNCH_KERNEL(529); break;
		case 530: LAUNCH_KERNEL(530); break;
		case 531: LAUNCH_KERNEL(531); break;
		case 532: LAUNCH_KERNEL(532); break;
		case 533: LAUNCH_KERNEL(533); break;
		case 534: LAUNCH_KERNEL(534); break;
		case 535: LAUNCH_KERNEL(535); break;
		case 536: LAUNCH_KERNEL(536); break;
		case 537: LAUNCH_KERNEL(537); break;
		case 538: LAUNCH_KERNEL(538); break;
		case 539: LAUNCH_KERNEL(539); break;
		case 540: LAUNCH_KERNEL(540); break;
		case 541: LAUNCH_KERNEL(541); break;
		case 542: LAUNCH_KERNEL(542); break;
		case 543: LAUNCH_KERNEL(543); break;
		case 544: LAUNCH_KERNEL(544); break;
		case 545: LAUNCH_KERNEL(545); break;
		case 546: LAUNCH_KERNEL(546); break;
		case 547: LAUNCH_KERNEL(547); break;
		case 548: LAUNCH_KERNEL(548); break;
		case 549: LAUNCH_KERNEL(549); break;
		case 550: LAUNCH_KERNEL(550); break;
		case 551: LAUNCH_KERNEL(551); break;
		case 552: LAUNCH_KERNEL(552); break;
		case 553: LAUNCH_KERNEL(553); break;
		case 554: LAUNCH_KERNEL(554); break;
		case 555: LAUNCH_KERNEL(555); break;
		case 556: LAUNCH_KERNEL(556); break;
		case 557: LAUNCH_KERNEL(557); break;
		case 558: LAUNCH_KERNEL(558); break;
		case 559: LAUNCH_KERNEL(559); break;
		case 560: LAUNCH_KERNEL(560); break;
		case 561: LAUNCH_KERNEL(561); break;
		case 562: LAUNCH_KERNEL(562); break;
		case 563: LAUNCH_KERNEL(563); break;
		case 564: LAUNCH_KERNEL(564); break;
		case 565: LAUNCH_KERNEL(565); break;
		case 566: LAUNCH_KERNEL(566); break;
		case 567: LAUNCH_KERNEL(567); break;
		case 568: LAUNCH_KERNEL(568); break;
		case 569: LAUNCH_KERNEL(569); break;
		case 570: LAUNCH_KERNEL(570); break;
		case 571: LAUNCH_KERNEL(571); break;
		case 572: LAUNCH_KERNEL(572); break;
		case 573: LAUNCH_KERNEL(573); break;
		case 574: LAUNCH_KERNEL(574); break;
		case 575: LAUNCH_KERNEL(575); break;
		case 576: LAUNCH_KERNEL(576); break;
		case 577: LAUNCH_KERNEL(577); break;
		case 578: LAUNCH_KERNEL(578); break;
		case 579: LAUNCH_KERNEL(579); break;
		case 580: LAUNCH_KERNEL(580); break;
		case 581: LAUNCH_KERNEL(581); break;
		case 582: LAUNCH_KERNEL(582); break;
		case 583: LAUNCH_KERNEL(583); break;
		case 584: LAUNCH_KERNEL(584); break;
		case 585: LAUNCH_KERNEL(585); break;
		case 586: LAUNCH_KERNEL(586); break;
		case 587: LAUNCH_KERNEL(587); break;
		case 588: LAUNCH_KERNEL(588); break;
		case 589: LAUNCH_KERNEL(589); break;
		case 590: LAUNCH_KERNEL(590); break;
		case 591: LAUNCH_KERNEL(591); break;
		case 592: LAUNCH_KERNEL(592); break;
		case 593: LAUNCH_KERNEL(593); break;
		case 594: LAUNCH_KERNEL(594); break;
		case 595: LAUNCH_KERNEL(595); break;
		case 596: LAUNCH_KERNEL(596); break;
		case 597: LAUNCH_KERNEL(597); break;
		case 598: LAUNCH_KERNEL(598); break;
		case 599: LAUNCH_KERNEL(599); break;
		case 600: LAUNCH_KERNEL(600); break;
		case 601: LAUNCH_KERNEL(601); break;
		case 602: LAUNCH_KERNEL(602); break;
		case 603: LAUNCH_KERNEL(603); break;
		case 604: LAUNCH_KERNEL(604); break;
		case 605: LAUNCH_KERNEL(605); break;
		case 606: LAUNCH_KERNEL(606); break;
		case 607: LAUNCH_KERNEL(607); break;
		case 608: LAUNCH_KERNEL(608); break;
		case 609: LAUNCH_KERNEL(609); break;
		case 610: LAUNCH_KERNEL(610); break;
		case 611: LAUNCH_KERNEL(611); break;
		case 612: LAUNCH_KERNEL(612); break;
		case 613: LAUNCH_KERNEL(613); break;
		case 614: LAUNCH_KERNEL(614); break;
		case 615: LAUNCH_KERNEL(615); break;
		case 616: LAUNCH_KERNEL(616); break;
		case 617: LAUNCH_KERNEL(617); break;
		case 618: LAUNCH_KERNEL(618); break;
		case 619: LAUNCH_KERNEL(619); break;
		case 620: LAUNCH_KERNEL(620); break;
		case 621: LAUNCH_KERNEL(621); break;
		case 622: LAUNCH_KERNEL(622); break;
		case 623: LAUNCH_KERNEL(623); break;
		case 624: LAUNCH_KERNEL(624); break;
		case 625: LAUNCH_KERNEL(625); break;
		case 626: LAUNCH_KERNEL(626); break;
		case 627: LAUNCH_KERNEL(627); break;
		case 628: LAUNCH_KERNEL(628); break;
		case 629: LAUNCH_KERNEL(629); break;
		case 630: LAUNCH_KERNEL(630); break;
		case 631: LAUNCH_KERNEL(631); break;
		case 632: LAUNCH_KERNEL(632); break;
		case 633: LAUNCH_KERNEL(633); break;
		case 634: LAUNCH_KERNEL(634); break;
		case 635: LAUNCH_KERNEL(635); break;
		case 636: LAUNCH_KERNEL(636); break;
		case 637: LAUNCH_KERNEL(637); break;
		case 638: LAUNCH_KERNEL(638); break;
		case 639: LAUNCH_KERNEL(639); break;
		case 640: LAUNCH_KERNEL(640); break;
		case 641: LAUNCH_KERNEL(641); break;
		case 642: LAUNCH_KERNEL(642); break;
		case 643: LAUNCH_KERNEL(643); break;
		case 644: LAUNCH_KERNEL(644); break;
		case 645: LAUNCH_KERNEL(645); break;
		case 646: LAUNCH_KERNEL(646); break;
		case 647: LAUNCH_KERNEL(647); break;
		case 648: LAUNCH_KERNEL(648); break;
		case 649: LAUNCH_KERNEL(649); break;
		case 650: LAUNCH_KERNEL(650); break;
		case 651: LAUNCH_KERNEL(651); break;
		case 652: LAUNCH_KERNEL(652); break;
		case 653: LAUNCH_KERNEL(653); break;
		case 654: LAUNCH_KERNEL(654); break;
		case 655: LAUNCH_KERNEL(655); break;
		case 656: LAUNCH_KERNEL(656); break;
		case 657: LAUNCH_KERNEL(657); break;
		case 658: LAUNCH_KERNEL(658); break;
		case 659: LAUNCH_KERNEL(659); break;
		case 660: LAUNCH_KERNEL(660); break;
		case 661: LAUNCH_KERNEL(661); break;
		case 662: LAUNCH_KERNEL(662); break;
		case 663: LAUNCH_KERNEL(663); break;
		case 664: LAUNCH_KERNEL(664); break;
		case 665: LAUNCH_KERNEL(665); break;
		case 666: LAUNCH_KERNEL(666); break;
		case 667: LAUNCH_KERNEL(667); break;
		case 668: LAUNCH_KERNEL(668); break;
		case 669: LAUNCH_KERNEL(669); break;
		case 670: LAUNCH_KERNEL(670); break;
		case 671: LAUNCH_KERNEL(671); break;
		case 672: LAUNCH_KERNEL(672); break;
		case 673: LAUNCH_KERNEL(673); break;
		case 674: LAUNCH_KERNEL(674); break;
		case 675: LAUNCH_KERNEL(675); break;
		case 676: LAUNCH_KERNEL(676); break;
		case 677: LAUNCH_KERNEL(677); break;
		case 678: LAUNCH_KERNEL(678); break;
		case 679: LAUNCH_KERNEL(679); break;
		case 680: LAUNCH_KERNEL(680); break;
		case 681: LAUNCH_KERNEL(681); break;
		case 682: LAUNCH_KERNEL(682); break;
		case 683: LAUNCH_KERNEL(683); break;
		case 684: LAUNCH_KERNEL(684); break;
		case 685: LAUNCH_KERNEL(685); break;
		case 686: LAUNCH_KERNEL(686); break;
		case 687: LAUNCH_KERNEL(687); break;
		case 688: LAUNCH_KERNEL(688); break;
		case 689: LAUNCH_KERNEL(689); break;
		case 690: LAUNCH_KERNEL(690); break;
		case 691: LAUNCH_KERNEL(691); break;
		case 692: LAUNCH_KERNEL(692); break;
		case 693: LAUNCH_KERNEL(693); break;
		case 694: LAUNCH_KERNEL(694); break;
		case 695: LAUNCH_KERNEL(695); break;
		case 696: LAUNCH_KERNEL(696); break;
		case 697: LAUNCH_KERNEL(697); break;
		case 698: LAUNCH_KERNEL(698); break;
		case 699: LAUNCH_KERNEL(699); break;
		case 700: LAUNCH_KERNEL(700); break;
		case 701: LAUNCH_KERNEL(701); break;
		case 702: LAUNCH_KERNEL(702); break;
		case 703: LAUNCH_KERNEL(703); break;
		case 704: LAUNCH_KERNEL(704); break;
		case 705: LAUNCH_KERNEL(705); break;
		case 706: LAUNCH_KERNEL(706); break;
		case 707: LAUNCH_KERNEL(707); break;
		case 708: LAUNCH_KERNEL(708); break;
		case 709: LAUNCH_KERNEL(709); break;
		case 710: LAUNCH_KERNEL(710); break;
		case 711: LAUNCH_KERNEL(711); break;
		case 712: LAUNCH_KERNEL(712); break;
		case 713: LAUNCH_KERNEL(713); break;
		case 714: LAUNCH_KERNEL(714); break;
		case 715: LAUNCH_KERNEL(715); break;
		case 716: LAUNCH_KERNEL(716); break;
		case 717: LAUNCH_KERNEL(717); break;
		case 718: LAUNCH_KERNEL(718); break;
		case 719: LAUNCH_KERNEL(719); break;
		case 720: LAUNCH_KERNEL(720); break;
		case 721: LAUNCH_KERNEL(721); break;
		case 722: LAUNCH_KERNEL(722); break;
		case 723: LAUNCH_KERNEL(723); break;
		case 724: LAUNCH_KERNEL(724); break;
		case 725: LAUNCH_KERNEL(725); break;
		case 726: LAUNCH_KERNEL(726); break;
		case 727: LAUNCH_KERNEL(727); break;
		case 728: LAUNCH_KERNEL(728); break;
		case 729: LAUNCH_KERNEL(729); break;
		case 730: LAUNCH_KERNEL(730); break;
		case 731: LAUNCH_KERNEL(731); break;
		case 732: LAUNCH_KERNEL(732); break;
		case 733: LAUNCH_KERNEL(733); break;
		case 734: LAUNCH_KERNEL(734); break;
		case 735: LAUNCH_KERNEL(735); break;
		case 736: LAUNCH_KERNEL(736); break;
		case 737: LAUNCH_KERNEL(737); break;
		case 738: LAUNCH_KERNEL(738); break;
		case 739: LAUNCH_KERNEL(739); break;
		case 740: LAUNCH_KERNEL(740); break;
		case 741: LAUNCH_KERNEL(741); break;
		case 742: LAUNCH_KERNEL(742); break;
		case 743: LAUNCH_KERNEL(743); break;
		case 744: LAUNCH_KERNEL(744); break;
		case 745: LAUNCH_KERNEL(745); break;
		case 746: LAUNCH_KERNEL(746); break;
		case 747: LAUNCH_KERNEL(747); break;
		case 748: LAUNCH_KERNEL(748); break;
		case 749: LAUNCH_KERNEL(749); break;
		case 750: LAUNCH_KERNEL(750); break;
		case 751: LAUNCH_KERNEL(751); break;
		case 752: LAUNCH_KERNEL(752); break;
		case 753: LAUNCH_KERNEL(753); break;
		case 754: LAUNCH_KERNEL(754); break;
		case 755: LAUNCH_KERNEL(755); break;
		case 756: LAUNCH_KERNEL(756); break;
		case 757: LAUNCH_KERNEL(757); break;
		case 758: LAUNCH_KERNEL(758); break;
		case 759: LAUNCH_KERNEL(759); break;
		case 760: LAUNCH_KERNEL(760); break;
		case 761: LAUNCH_KERNEL(761); break;
		case 762: LAUNCH_KERNEL(762); break;
		case 763: LAUNCH_KERNEL(763); break;
		case 764: LAUNCH_KERNEL(764); break;
		case 765: LAUNCH_KERNEL(765); break;
		case 766: LAUNCH_KERNEL(766); break;
		case 767: LAUNCH_KERNEL(767); break;
		case 768: LAUNCH_KERNEL(768); break;
		case 769: LAUNCH_KERNEL(769); break;
		case 770: LAUNCH_KERNEL(770); break;
		case 771: LAUNCH_KERNEL(771); break;
		case 772: LAUNCH_KERNEL(772); break;
		case 773: LAUNCH_KERNEL(773); break;
		case 774: LAUNCH_KERNEL(774); break;
		case 775: LAUNCH_KERNEL(775); break;
		case 776: LAUNCH_KERNEL(776); break;
		case 777: LAUNCH_KERNEL(777); break;
		case 778: LAUNCH_KERNEL(778); break;
		case 779: LAUNCH_KERNEL(779); break;
		case 780: LAUNCH_KERNEL(780); break;
		case 781: LAUNCH_KERNEL(781); break;
		case 782: LAUNCH_KERNEL(782); break;
		case 783: LAUNCH_KERNEL(783); break;
		case 784: LAUNCH_KERNEL(784); break;
		case 785: LAUNCH_KERNEL(785); break;
		case 786: LAUNCH_KERNEL(786); break;
		case 787: LAUNCH_KERNEL(787); break;
		case 788: LAUNCH_KERNEL(788); break;
		case 789: LAUNCH_KERNEL(789); break;
		case 790: LAUNCH_KERNEL(790); break;
		case 791: LAUNCH_KERNEL(791); break;
		case 792: LAUNCH_KERNEL(792); break;
		case 793: LAUNCH_KERNEL(793); break;
		case 794: LAUNCH_KERNEL(794); break;
		case 795: LAUNCH_KERNEL(795); break;
		case 796: LAUNCH_KERNEL(796); break;
		case 797: LAUNCH_KERNEL(797); break;
		case 798: LAUNCH_KERNEL(798); break;
		case 799: LAUNCH_KERNEL(799); break;
		case 800: LAUNCH_KERNEL(800); break;
		case 801: LAUNCH_KERNEL(801); break;
		case 802: LAUNCH_KERNEL(802); break;
		case 803: LAUNCH_KERNEL(803); break;
		case 804: LAUNCH_KERNEL(804); break;
		case 805: LAUNCH_KERNEL(805); break;
		case 806: LAUNCH_KERNEL(806); break;
		case 807: LAUNCH_KERNEL(807); break;
		case 808: LAUNCH_KERNEL(808); break;
		case 809: LAUNCH_KERNEL(809); break;
		case 810: LAUNCH_KERNEL(810); break;
		case 811: LAUNCH_KERNEL(811); break;
		case 812: LAUNCH_KERNEL(812); break;
		case 813: LAUNCH_KERNEL(813); break;
		case 814: LAUNCH_KERNEL(814); break;
		case 815: LAUNCH_KERNEL(815); break;
		case 816: LAUNCH_KERNEL(816); break;
		case 817: LAUNCH_KERNEL(817); break;
		case 818: LAUNCH_KERNEL(818); break;
		case 819: LAUNCH_KERNEL(819); break;
		case 820: LAUNCH_KERNEL(820); break;
		case 821: LAUNCH_KERNEL(821); break;
		case 822: LAUNCH_KERNEL(822); break;
		case 823: LAUNCH_KERNEL(823); break;
		case 824: LAUNCH_KERNEL(824); break;
		case 825: LAUNCH_KERNEL(825); break;
		case 826: LAUNCH_KERNEL(826); break;
		case 827: LAUNCH_KERNEL(827); break;
		case 828: LAUNCH_KERNEL(828); break;
		case 829: LAUNCH_KERNEL(829); break;
		case 830: LAUNCH_KERNEL(830); break;
		case 831: LAUNCH_KERNEL(831); break;
		case 832: LAUNCH_KERNEL(832); break;
		case 833: LAUNCH_KERNEL(833); break;
		case 834: LAUNCH_KERNEL(834); break;
		case 835: LAUNCH_KERNEL(835); break;
		case 836: LAUNCH_KERNEL(836); break;
		case 837: LAUNCH_KERNEL(837); break;
		case 838: LAUNCH_KERNEL(838); break;
		case 839: LAUNCH_KERNEL(839); break;
		case 840: LAUNCH_KERNEL(840); break;
		case 841: LAUNCH_KERNEL(841); break;
		case 842: LAUNCH_KERNEL(842); break;
		case 843: LAUNCH_KERNEL(843); break;
		case 844: LAUNCH_KERNEL(844); break;
		case 845: LAUNCH_KERNEL(845); break;
		case 846: LAUNCH_KERNEL(846); break;
		case 847: LAUNCH_KERNEL(847); break;
		case 848: LAUNCH_KERNEL(848); break;
		case 849: LAUNCH_KERNEL(849); break;
		case 850: LAUNCH_KERNEL(850); break;
		case 851: LAUNCH_KERNEL(851); break;
		case 852: LAUNCH_KERNEL(852); break;
		case 853: LAUNCH_KERNEL(853); break;
		case 854: LAUNCH_KERNEL(854); break;
		case 855: LAUNCH_KERNEL(855); break;
		case 856: LAUNCH_KERNEL(856); break;
		case 857: LAUNCH_KERNEL(857); break;
		case 858: LAUNCH_KERNEL(858); break;
		case 859: LAUNCH_KERNEL(859); break;
		case 860: LAUNCH_KERNEL(860); break;
		case 861: LAUNCH_KERNEL(861); break;
		case 862: LAUNCH_KERNEL(862); break;
		case 863: LAUNCH_KERNEL(863); break;
		case 864: LAUNCH_KERNEL(864); break;
		case 865: LAUNCH_KERNEL(865); break;
		case 866: LAUNCH_KERNEL(866); break;
		case 867: LAUNCH_KERNEL(867); break;
		case 868: LAUNCH_KERNEL(868); break;
		case 869: LAUNCH_KERNEL(869); break;
		case 870: LAUNCH_KERNEL(870); break;
		case 871: LAUNCH_KERNEL(871); break;
		case 872: LAUNCH_KERNEL(872); break;
		case 873: LAUNCH_KERNEL(873); break;
		case 874: LAUNCH_KERNEL(874); break;
		case 875: LAUNCH_KERNEL(875); break;
		case 876: LAUNCH_KERNEL(876); break;
		case 877: LAUNCH_KERNEL(877); break;
		case 878: LAUNCH_KERNEL(878); break;
		case 879: LAUNCH_KERNEL(879); break;
		case 880: LAUNCH_KERNEL(880); break;
		case 881: LAUNCH_KERNEL(881); break;
		case 882: LAUNCH_KERNEL(882); break;
		case 883: LAUNCH_KERNEL(883); break;
		case 884: LAUNCH_KERNEL(884); break;
		case 885: LAUNCH_KERNEL(885); break;
		case 886: LAUNCH_KERNEL(886); break;
		case 887: LAUNCH_KERNEL(887); break;
		case 888: LAUNCH_KERNEL(888); break;
		case 889: LAUNCH_KERNEL(889); break;
		case 890: LAUNCH_KERNEL(890); break;
		case 891: LAUNCH_KERNEL(891); break;
		case 892: LAUNCH_KERNEL(892); break;
		case 893: LAUNCH_KERNEL(893); break;
		case 894: LAUNCH_KERNEL(894); break;
		case 895: LAUNCH_KERNEL(895); break;
		case 896: LAUNCH_KERNEL(896); break;
		case 897: LAUNCH_KERNEL(897); break;
		case 898: LAUNCH_KERNEL(898); break;
		case 899: LAUNCH_KERNEL(899); break;
		case 900: LAUNCH_KERNEL(900); break;
		case 901: LAUNCH_KERNEL(901); break;
		case 902: LAUNCH_KERNEL(902); break;
		case 903: LAUNCH_KERNEL(903); break;
		case 904: LAUNCH_KERNEL(904); break;
		case 905: LAUNCH_KERNEL(905); break;
		case 906: LAUNCH_KERNEL(906); break;
		case 907: LAUNCH_KERNEL(907); break;
		case 908: LAUNCH_KERNEL(908); break;
		case 909: LAUNCH_KERNEL(909); break;
		case 910: LAUNCH_KERNEL(910); break;
		case 911: LAUNCH_KERNEL(911); break;
		case 912: LAUNCH_KERNEL(912); break;
		case 913: LAUNCH_KERNEL(913); break;
		case 914: LAUNCH_KERNEL(914); break;
		case 915: LAUNCH_KERNEL(915); break;
		case 916: LAUNCH_KERNEL(916); break;
		case 917: LAUNCH_KERNEL(917); break;
		case 918: LAUNCH_KERNEL(918); break;
		case 919: LAUNCH_KERNEL(919); break;
		case 920: LAUNCH_KERNEL(920); break;
		case 921: LAUNCH_KERNEL(921); break;
		case 922: LAUNCH_KERNEL(922); break;
		case 923: LAUNCH_KERNEL(923); break;
		case 924: LAUNCH_KERNEL(924); break;
		case 925: LAUNCH_KERNEL(925); break;
		case 926: LAUNCH_KERNEL(926); break;
		case 927: LAUNCH_KERNEL(927); break;
		case 928: LAUNCH_KERNEL(928); break;
		case 929: LAUNCH_KERNEL(929); break;
		case 930: LAUNCH_KERNEL(930); break;
		case 931: LAUNCH_KERNEL(931); break;
		case 932: LAUNCH_KERNEL(932); break;
		case 933: LAUNCH_KERNEL(933); break;
		case 934: LAUNCH_KERNEL(934); break;
		case 935: LAUNCH_KERNEL(935); break;
		case 936: LAUNCH_KERNEL(936); break;
		case 937: LAUNCH_KERNEL(937); break;
		case 938: LAUNCH_KERNEL(938); break;
		case 939: LAUNCH_KERNEL(939); break;
		case 940: LAUNCH_KERNEL(940); break;
		case 941: LAUNCH_KERNEL(941); break;
		case 942: LAUNCH_KERNEL(942); break;
		case 943: LAUNCH_KERNEL(943); break;
		case 944: LAUNCH_KERNEL(944); break;
		case 945: LAUNCH_KERNEL(945); break;
		case 946: LAUNCH_KERNEL(946); break;
		case 947: LAUNCH_KERNEL(947); break;
		case 948: LAUNCH_KERNEL(948); break;
		case 949: LAUNCH_KERNEL(949); break;
		case 950: LAUNCH_KERNEL(950); break;
		case 951: LAUNCH_KERNEL(951); break;
		case 952: LAUNCH_KERNEL(952); break;
		case 953: LAUNCH_KERNEL(953); break;
		case 954: LAUNCH_KERNEL(954); break;
		case 955: LAUNCH_KERNEL(955); break;
		case 956: LAUNCH_KERNEL(956); break;
		case 957: LAUNCH_KERNEL(957); break;
		case 958: LAUNCH_KERNEL(958); break;
		case 959: LAUNCH_KERNEL(959); break;
		case 960: LAUNCH_KERNEL(960); break;
		case 961: LAUNCH_KERNEL(961); break;
		case 962: LAUNCH_KERNEL(962); break;
		case 963: LAUNCH_KERNEL(963); break;
		case 964: LAUNCH_KERNEL(964); break;
		case 965: LAUNCH_KERNEL(965); break;
		case 966: LAUNCH_KERNEL(966); break;
		case 967: LAUNCH_KERNEL(967); break;
		case 968: LAUNCH_KERNEL(968); break;
		case 969: LAUNCH_KERNEL(969); break;
		case 970: LAUNCH_KERNEL(970); break;
		case 971: LAUNCH_KERNEL(971); break;
		case 972: LAUNCH_KERNEL(972); break;
		case 973: LAUNCH_KERNEL(973); break;
		case 974: LAUNCH_KERNEL(974); break;
		case 975: LAUNCH_KERNEL(975); break;
		case 976: LAUNCH_KERNEL(976); break;
		case 977: LAUNCH_KERNEL(977); break;
		case 978: LAUNCH_KERNEL(978); break;
		case 979: LAUNCH_KERNEL(979); break;
		case 980: LAUNCH_KERNEL(980); break;
		case 981: LAUNCH_KERNEL(981); break;
		case 982: LAUNCH_KERNEL(982); break;
		case 983: LAUNCH_KERNEL(983); break;
		case 984: LAUNCH_KERNEL(984); break;
		case 985: LAUNCH_KERNEL(985); break;
		case 986: LAUNCH_KERNEL(986); break;
		case 987: LAUNCH_KERNEL(987); break;
		case 988: LAUNCH_KERNEL(988); break;
		case 989: LAUNCH_KERNEL(989); break;
		case 990: LAUNCH_KERNEL(990); break;
		case 991: LAUNCH_KERNEL(991); break;
		case 992: LAUNCH_KERNEL(992); break;
		case 993: LAUNCH_KERNEL(993); break;
		case 994: LAUNCH_KERNEL(994); break;
		case 995: LAUNCH_KERNEL(995); break;
		case 996: LAUNCH_KERNEL(996); break;
		case 997: LAUNCH_KERNEL(997); break;
		case 998: LAUNCH_KERNEL(998); break;
		case 999: LAUNCH_KERNEL(999); break;
		case 1000: LAUNCH_KERNEL(1000); break;
		case 1001: LAUNCH_KERNEL(1001); break;
		case 1002: LAUNCH_KERNEL(1002); break;
		case 1003: LAUNCH_KERNEL(1003); break;
		case 1004: LAUNCH_KERNEL(1004); break;
		case 1005: LAUNCH_KERNEL(1005); break;
		case 1006: LAUNCH_KERNEL(1006); break;
		case 1007: LAUNCH_KERNEL(1007); break;
		case 1008: LAUNCH_KERNEL(1008); break;
		case 1009: LAUNCH_KERNEL(1009); break;
		case 1010: LAUNCH_KERNEL(1010); break;
		case 1011: LAUNCH_KERNEL(1011); break;
		case 1012: LAUNCH_KERNEL(1012); break;
		case 1013: LAUNCH_KERNEL(1013); break;
		case 1014: LAUNCH_KERNEL(1014); break;
		case 1015: LAUNCH_KERNEL(1015); break;
		case 1016: LAUNCH_KERNEL(1016); break;
		case 1017: LAUNCH_KERNEL(1017); break;
		case 1018: LAUNCH_KERNEL(1018); break;
		case 1019: LAUNCH_KERNEL(1019); break;
		case 1020: LAUNCH_KERNEL(1020); break;
		case 1021: LAUNCH_KERNEL(1021); break;
		case 1022: LAUNCH_KERNEL(1022); break;
		case 1023: LAUNCH_KERNEL(1023); break;
		default: ASSERT(FALSE);
	}
}

#endif
