#include "hip/hip_runtime.h"
// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



// TO DO: Use smallKeyBitmap[]!



///////////////////////////////////////////////////////////////////////////////
// INCLUDE FILE(S)                                                           //
///////////////////////////////////////////////////////////////////////////////

#include "MerikensTripcodeEngine.h"



///////////////////////////////////////////////////////////////////////////////
// VARIABLES FOR CUDA CODES                                                  //
///////////////////////////////////////////////////////////////////////////////

__device__ __constant__ unsigned char   CUDA_keyCharTable_OneByte[SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   CUDA_keyCharTable_FirstByte  [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   CUDA_keyCharTable_SecondByte [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ char            CUDA_base64CharTable[64];
__device__ __constant__ unsigned char   CUDA_key[12];



///////////////////////////////////////////////////////////////////////////////
// BITSLICE DES                                                              //
///////////////////////////////////////////////////////////////////////////////

#define CUDA_DES_BS_DEPTH                   32
#define CUDA_DES_NUM_THREADS_PER_BLOCK      64
#define CUDA_DES_NUM_THREADS_FOR_BITSLICE   1
#define CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK (CUDA_DES_NUM_THREADS_PER_BLOCK / CUDA_DES_NUM_THREADS_FOR_BITSLICE)

typedef int           DES_ARCH_WORD;
typedef int           DES_ARCH_WORD_32;
#define DES_ARCH_SIZE 4
#define DES_ARCH_BITS 32

typedef int           DES_Vector;
// #define CUDA_DES_BS_DEPTH  DES_ARCH_BITS
#define DES_VECTOR_ZERO               0
#define DES_VECTOR_ONES               ~(DES_Vector)0

#define DES_VECTOR_NOT(dst, a)        (dst) =  ~(a)
#define DES_VECTOR_AND(dst, a, b)     (dst) =   (a) &  (b)
#define DES_VECTOR_OR(dst, a, b)      (dst) =   (a) |  (b)
#define DES_VECTOR_AND_NOT(dst, a, b) (dst) =   (a) & ~(b)
#define DES_VECTOR_XOR_NOT(dst, a, b) (dst) = ~((a) ^  (b))
#define DES_VECTOR_NOT_OR(dst, a, b)  (dst) = ~((a) |  (b))
#define DES_VECTOR_SEL(dst, a, b, c)  (dst) = (((a) & ~(c)) ^ ((b) & (c)))
#define DES_VECTOR_XOR_FUNC(a, b)              ((a) ^  (b))
#define DES_VECTOR_XOR(dst, a, b)     (dst) = DES_VECTOR_XOR_FUNC((a), (b))
#define DES_VECTOR_SET(dst, ofs, src) *((DES_Vector *)((DES_Vector *)&(dst) + ((ofs) << DB_SHIFT))) = (src)

#define DES_CONSTANT_QUALIFIERS      __device__ __constant__
#define DES_FUNCTION_QUALIFIERS      __device__ __forceinline__
#define DES_SBOX_FUNCTION_QUALIFIERS __device__ __forceinline__

__device__ __shared__ DES_Vector dataBlocks[64 * CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK];
#define DB_SHIFT 6

const unsigned char expansionTable[48] = {
	31,  0,  1,  2,  3,  4,
	 3,  4,  5,  6,  7,  8,
	 7,  8,  9, 10, 11, 12,
	11, 12, 13, 14, 15, 16,
	15, 16, 17, 18, 19, 20,
	19, 20, 21, 22, 23, 24,
	23, 24, 25, 26, 27, 28,
	27, 28, 29, 30, 31,  0
};

__device__ __constant__ unsigned char CUDA_expansionFunction[96];
__device__ __constant__ unsigned char CUDA_key7Array[CUDA_DES_BS_DEPTH];
__device__ __constant__ DES_Vector    CUDA_keyFrom49To55Array[7];

const char charToIndexTableForDES[0x100] = {
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x00, 0x01,
	0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09,
	0x0a, 0x0b, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x0c, 0x0d, 0x0e, 0x0f, 0x10, 0x11, 0x12,
	0x13, 0x14, 0x15, 0x16, 0x17, 0x18, 0x19, 0x1a,
	0x1b, 0x1c, 0x1d, 0x1e, 0x1f, 0x20, 0x21, 0x22,
	0x23, 0x24, 0x25, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x26, 0x27, 0x28, 0x29, 0x2a, 0x2b, 0x2c,
	0x2d, 0x2e, 0x2f, 0x30, 0x31, 0x32, 0x33, 0x34,
	0x35, 0x36, 0x37, 0x38, 0x39, 0x3a, 0x3b, 0x3c,
	0x3d, 0x3e, 0x3f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
};

DES_CONSTANT_QUALIFIERS char CUDA_DES_indexToCharTable[64] =
//	"./0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
{
	/* 00 */ '.', '/',
	/* 02 */ '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 
	/* 12 */ 'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 
	/* 28 */ 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
	/* 38 */ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p',
	/* 54 */ 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z', 
};

DES_CONSTANT_QUALIFIERS unsigned char keySchedule[DES_SIZE_KEY_SCHEDULE] = {
	12, 46, 33, 52, 48, 20, 34, 55,  5, 13, 18, 40,  4, 32, 26, 27,
	38, 54, 53,  6, 31, 25, 19, 41, 15, 24, 28, 43, 30,  3, 35, 22,
	 2, 44, 14, 23, 51, 16, 29, 49,  7, 17, 37,  8,  9, 50, 42, 21,
	 5, 39, 26, 45, 41, 13, 27, 48, 53,  6, 11, 33, 52, 25, 19, 20,
	31, 47, 46, 54, 55, 18, 12, 34,  8, 17, 21, 36, 23, 49, 28, 15,
	24, 37,  7, 16, 44,  9, 22, 42,  0, 10, 30,  1,  2, 43, 35, 14,
	46, 25, 12, 31, 27, 54, 13, 34, 39, 47, 52, 19, 38, 11,  5,  6,
	48, 33, 32, 40, 41,  4, 53, 20, 51,  3,  7, 22,  9, 35, 14,  1,
	10, 23, 50,  2, 30, 24,  8, 28, 43, 49, 16, 44, 17, 29, 21,  0,
	32, 11, 53, 48, 13, 40, 54, 20, 25, 33, 38,  5, 55, 52, 46, 47,
	34, 19, 18, 26, 27, 45, 39,  6, 37, 42, 50,  8, 24, 21,  0, 44,
	49,  9, 36, 17, 16, 10, 51, 14, 29, 35,  2, 30,  3, 15,  7, 43,
	18, 52, 39, 34, 54, 26, 40,  6, 11, 19, 55, 46, 41, 38, 32, 33,
	20,  5,  4, 12, 13, 31, 25, 47, 23, 28, 36, 51, 10,  7, 43, 30,
	35, 24, 22,  3,  2, 49, 37,  0, 15, 21, 17, 16, 42,  1, 50, 29,
	 4, 38, 25, 20, 40, 12, 26, 47, 52,  5, 41, 32, 27, 55, 18, 19,
	 6, 46, 45, 53, 54, 48, 11, 33,  9, 14, 22, 37, 49, 50, 29, 16,
	21, 10,  8, 42, 17, 35, 23, 43,  1,  7,  3,  2, 28, 44, 36, 15,
	45, 55, 11,  6, 26, 53, 12, 33, 38, 46, 27, 18, 13, 41,  4,  5,
	47, 32, 31, 39, 40, 34, 52, 19, 24,  0,  8, 23, 35, 36, 15,  2,
	 7, 49, 51, 28,  3, 21,  9, 29, 44, 50, 42, 17, 14, 30, 22,  1,
	31, 41, 52, 47, 12, 39, 53, 19, 55, 32, 13,  4, 54, 27, 45, 46,
	33, 18, 48, 25, 26, 20, 38,  5, 10, 43, 51,  9, 21, 22,  1, 17,
	50, 35, 37, 14, 42,  7, 24, 15, 30, 36, 28,  3,  0, 16,  8, 44,
	55, 34, 45, 40,  5, 32, 46, 12, 48, 25,  6, 52, 47, 20, 38, 39,
	26, 11, 41, 18, 19, 13, 31, 53,  3, 36, 44,  2, 14, 15, 51, 10,
	43, 28, 30,  7, 35,  0, 17,  8, 23, 29, 21, 49, 50,  9,  1, 37,
	41, 20, 31, 26, 46, 18, 32, 53, 34, 11, 47, 38, 33,  6, 55, 25,
	12, 52, 27,  4,  5, 54, 48, 39, 42, 22, 30, 17,  0,  1, 37, 49,
	29, 14, 16, 50, 21, 43,  3, 51,  9, 15,  7, 35, 36, 24, 44, 23,
	27,  6, 48, 12, 32,  4, 18, 39, 20, 52, 33, 55, 19, 47, 41, 11,
	53, 38, 13, 45, 46, 40, 34, 25, 28,  8, 16,  3, 43, 44, 23, 35,
	15,  0,  2, 36,  7, 29, 42, 37, 24,  1, 50, 21, 22, 10, 30,  9,
	13, 47, 34, 53, 18, 45,  4, 25,  6, 38, 19, 41,  5, 33, 27, 52,
	39, 55, 54, 31, 32, 26, 20, 11, 14, 51,  2, 42, 29, 30,  9, 21,
	 1, 43, 17, 22, 50, 15, 28, 23, 10, 44, 36,  7,  8, 49, 16, 24,
	54, 33, 20, 39,  4, 31, 45, 11, 47, 55,  5, 27, 46, 19, 13, 38,
	25, 41, 40, 48, 18, 12,  6, 52,  0, 37, 17, 28, 15, 16, 24,  7,
	44, 29,  3,  8, 36,  1, 14,  9, 49, 30, 22, 50, 51, 35,  2, 10,
	40, 19,  6, 25, 45, 48, 31, 52, 33, 41, 46, 13, 32,  5, 54, 55,
	11, 27, 26, 34,  4, 53, 47, 38, 43, 23,  3, 14,  1,  2, 10, 50,
	30, 15, 42, 51, 22, 44,  0, 24, 35, 16,  8, 36, 37, 21, 17, 49,
	26,  5, 47, 11, 31, 34, 48, 38, 19, 27, 32, 54, 18, 46, 40, 41,
	52, 13, 12, 20, 45, 39, 33, 55, 29,  9, 42,  0, 44, 17, 49, 36,
	16,  1, 28, 37,  8, 30, 43, 10, 21,  2, 51, 22, 23,  7,  3, 35,
	19, 53, 40,  4, 55, 27, 41, 31, 12, 20, 25, 47, 11, 39, 33, 34,
	45,  6,  5, 13, 38, 32, 26, 48, 22,  2, 35, 50, 37, 10, 42, 29,
	 9, 51, 21, 30,  1, 23, 36,  3, 14, 24, 44, 15, 16,  0, 49, 28,
};

void DES_CreateExpansionFunction(char *saltString, unsigned char *expansionFunction)
{
	unsigned char saltChar1 = '.', saltChar2 = '.';
	DES_ARCH_WORD salt;
	DES_ARCH_WORD mask;
	int src, dst;

	if (saltString[0]) {
		saltChar1 = saltString[0];
		if (saltString[1])
			saltChar2 = saltString[1];
	}
	salt =    charToIndexTableForDES[saltChar1]
	       | (charToIndexTableForDES[saltChar2] << 6);

	mask = 1;
	for (dst = 0; dst < 48; dst++) {
		if (dst == 24) mask = 1;

		if (salt & mask) {
			if (dst < 24) src = dst + 24; else src = dst - 24;
		} else src = dst;

		expansionFunction[dst     ] = expansionTable[src];
		expansionFunction[dst + 48] = expansionTable[src] + 32;

		mask <<= 1;
	}
}

// Bitslice DES S-boxes for x86 with MMX/SSE2/AVX and for typical RISC
// architectures.  These use AND, OR, XOR, NOT, and AND-NOT gates.
//
// Gate counts: 49 44 46 33 48 46 46 41
// Average: 44.125
//
// Several same-gate-count expressions for each S-box are included (for use on
// different CPUs/GPUs).
//
// These Boolean expressions corresponding to DES S-boxes have been generated
// by Roman Rusakov <roman_rus at openwall.com> for use in Openwall's
// John the Ripper password cracker: http://www.openwall.com/john/
// Being mathematical formulas, they are not copyrighted and are free for reuse
// by anyone.
//
// This file (a specific representation of the S-box expressions, surrounding
// logic) is Copyright (c) 2011 by Solar Designer <solar at openwall.com>.
// Redistribution and use in source and binary forms, with or without
// modification, are permitted.  (This is a heavily cut-down "BSD license".)
//
// The effort has been sponsored by Rapid7: http://www.rapid7.com

//
// s1-00484, 49 gates, 17 regs, 11 andn, 4/9/39/79/120 stalls, 74 biop
// Currently used for MMX/SSE2 and x86-64 SSE2
//
#define s1(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t0) = ~(a5);\
	(t0) = (a1) & (t0);\
	(t1) = (a4) ^ (t0);\
	(t2) = (a3) | (a6);\
	(t3) = (a1) ^ (a3);\
	(t4) = (t2) & (t3);\
	(t5) = (a4) ^ (t4);\
	(t6) = ~(t1);\
	(t6) = (t5) & (t6);\
	(t7) = (a5) ^ (a6);\
	(t8) = (a3) ^ (t7);\
	(t9) = ~(t8);\
	(t9) = (t1) & (t9);\
	(t8) = (a6) | (t4);\
	(t4) = (t9) ^ (t8);\
	(t8) = ~(t6);\
	(t8) = (t4) & (t8);\
	(t9) = (a1) | (a6);\
	(t10) = (t4) | (t9);\
	(t11) = ~(t5);\
	(t11) = (a5) & (t11);\
	(t5) = (t10) ^ (t11);\
	(t12) = ~(t9);\
	(t12) = (a4) & (t12);\
	(t9) = (t11) ^ (t12);\
	(t12) = ~(t3);\
	(t12) = (t7) & (t12);\
	(t3) = (t9) | (t12);\
	(t12) = ~(t0);\
	(t12) = (a3) & (t12);\
	(t0) = (t1) ^ (t10);\
	(t9) = ~(t12);\
	(t9) = (t0) & (t9);\
	(t12) = ~(t9);\
	(t0) = (t2) & (t4);\
	(t4) = (t12) ^ (t0);\
	(t13) = ~(a2);\
	(t13) = (t5) & (t13);\
	(t5) = (t13) ^ (t4);\
	(out3) = (out3) ^ (t5);\
	(t12) = (t7) ^ (t9);\
	(t0) = (t11) | (t12);\
	(t5) = (t2) ^ (t0);\
	(t11) = (a1) ^ (t5);\
	(t5) = (t4) ^ (t11);\
	(t9) = (t6) | (a2);\
	(t12) = (t9) ^ (t5);\
	(out1) = (out1) ^ (t12);\
	(t13) = (t2) ^ (t10);\
	(t0) = (t3) | (t13);\
	(t13) = (t11) ^ (t0);\
	(t0) = (t7) | (t5);\
	(t5) = (t13) ^ (t0);\
	(t0) = (t8) | (a2);\
	(t6) = (t0) ^ (t5);\
	(out2) = (out2) ^ (t6);\
	(t6) = (a5) | (t1);\
	(t9) = ~(t13);\
	(t9) = (t6) & (t9);\
	(t13) = (t8) & (t11);\
	(t11) = (t9) ^ (t13);\
	(t13) = (t11) | (a2);\
	(t12) = (t13) ^ (t3);\
	(out4) = (out4) ^ (t12);\

//
// s2-016251, 44 gates, 14 regs, 13 andn, 1/9/22/61/108 stalls, 66 biop */
//
#define s2(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t0) = (a2) ^ (a5);\
	(t1) = ~(a6);\
	(t1) = (a1) & (t1);\
	(t2) = ~(t1);\
	(t2) = (a5) & (t2);\
	(t1) = (a2) | (t2);\
	(t3) = ~(a6);\
	(t3) = (t0) & (t3);\
	(t4) = (a1) & (t0);\
	(t5) = (a5) ^ (t4);\
	(t6) = ~(t3);\
	(t6) = (t5) & (t6);\
	(t7) = (a3) & (a6);\
	(t8) = (t2) ^ (t3);\
	(t2) = (t1) & (t8);\
	(t3) = ~(t7);\
	(t3) = (t2) & (t3);\
	(t8) = (a3) & (t2);\
	(t2) = ~(a1);\
	(t9) = (t8) ^ (t2);\
	(t2) = (a6) ^ (t0);\
	(t0) = ~(t7);\
	(t0) = (t2) & (t0);\
	(t10) = (t9) ^ (t0);\
	(t11) = ~(t3);\
	(t11) = (a4) & (t11);\
	(t3) = (t11) ^ (t10);\
	(out2) = (out2) ^ (t3);\
	(t3) = ~(t0);\
	(t3) = (a2) & (t3);\
	(t0) = (t5) ^ (t3);\
	(t5) = ~(t0);\
	(t5) = (t9) & (t5);\
	(t9) = (a3) ^ (t2);\
	(t11) = (t5) ^ (t9);\
	(t5) = ~(a4);\
	(t5) = (t1) & (t5);\
	(t12) = (t5) ^ (t11);\
	(out1) = (out1) ^ (t12);\
	(t5) = (t8) ^ (t3);\
	(t3) = (t9) | (t5);\
	(t8) = (t1) ^ (t10);\
	(t1) = (t7) | (t8);\
	(t7) = (t3) ^ (t1);\
	(t1) = ~(t11);\
	(t1) = (t10) & (t1);\
	(t3) = (t4) ^ (t5);\
	(t4) = (t1) | (t3);\
	(t1) = ~(t9);\
	(t1) = (t6) & (t1);\
	(t3) = (t4) ^ (t1);\
	(t1) = (t3) | (a4);\
	(t4) = (t1) ^ (t7);\
	(out3) = (out3) ^ (t4);\
	(t1) = ~(t0);\
	(t1) = (t3) & (t1);\
	(t0) = (t2) | (t8);\
	(t2) = (t1) ^ (t0);\
	(t0) = (t6) | (a4);\
	(t1) = (t0) ^ (t2);\
	(out4) = (out4) ^ (t1);\

//
// s3-000426, 46 gates, 16 regs, 14 andn, 2/5/12/35/75 stalls, 68 biop
// Currently used for x86-64 SSE2
//
#define s3(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t0) = ~(a2);\
	(t0) = (a1) & (t0);\
	(t1) = (a3) ^ (a6);\
	(t2) = (t0) | (t1);\
	(t0) = (a4) ^ (a6);\
	(t3) = ~(a1);\
	(t3) = (t0) & (t3);\
	(t4) = (t2) ^ (t3);\
	(t5) = (a2) ^ (t1);\
	(t6) = ~(a6);\
	(t6) = (t5) & (t6);\
	(t7) = (t2) ^ (t6);\
	(t2) = ~(t7);\
	(t2) = (t4) & (t2);\
	(t6) = (a6) & (t4);\
	(t8) = (a4) | (t6);\
	(t6) = (a1) & (t8);\
	(t8) = (t5) ^ (t6);\
	(t6) = ~(a5);\
	(t6) = (t4) & (t6);\
	(t9) = (t6) ^ (t8);\
	(out4) = (out4) ^ (t9);\
	(t6) = (t1) & (t0);\
	(t0) = (a1) ^ (a4);\
	(t9) = (t7) ^ (t0);\
	(t7) = (a3) | (t9);\
	(t9) = ~(t6);\
	(t9) = (t7) & (t9);\
	(t6) = (t3) | (t0);\
	(t0) = ~(t6);\
	(t0) = (t8) & (t0);\
	(t7) = (a4) & (a6);\
	(t8) = ~(a2);\
	(t8) = (t7) & (t8);\
	(t10) = (t0) ^ (t8);\
	(t0) = ~(a3);\
	(t0) = (t10) & (t0);\
	(t8) = (t5) | (t7);\
	(t7) = ~(t0);\
	(t7) = (t8) & (t7);\
	(t0) = (a1) ^ (t7);\
	(t7) = (t9) & (a5);\
	(t8) = (t7) ^ (t0);\
	(out2) = (out2) ^ (t8);\
	(t0) = ~(a2);\
	(t0) = (t4) & (t0);\
	(t4) = ~(a3);\
	(t4) = (t0) & (t4);\
	(t7) = (t5) ^ (t6);\
	(t6) = ~(t7);\
	(t7) = (t4) ^ (t6);\
	(t4) = ~(t2);\
	(t4) = (a5) & (t4);\
	(t2) = (t4) ^ (t7);\
	(out1) = (out1) ^ (t2);\
	(t2) = (a4) & (t1);\
	(t1) = (t5) | (t7);\
	(t4) = ~(t2);\
	(t4) = (t1) & (t4);\
	(t1) = (t3) | (t0);\
	(t0) = (t4) ^ (t1);\
	(t1) = (t10) | (a5);\
	(t2) = (t1) ^ (t0);\
	(out3) = (out3) ^ (t2);\

//
// s4, 33 gates, 11/12 regs, 9 andn, 2/21/53/86/119 stalls, 52 biop
//
#define s4(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t0) = (a1) ^ (a3);\
	(t1) = (a3) ^ (a5);\
	(t2) = (a2) | (a4);\
	(t3) = (a5) ^ (t2);\
	(t2) = ~(t3);\
	(t2) = (t1) & (t2);\
	(t3) = ~(a2);\
	(t3) = (t1) & (t3);\
	(t4) = (a4) ^ (t3);\
	(t5) = (t0) | (t4);\
	(t6) = ~(t2);\
	(t6) = (t5) & (t6);\
	(t5) = (a2) ^ (t6);\
	(t7) = (t4) & (t5);\
	(t4) = ~(t7);\
	(t4) = (t1) & (t4);\
	(t1) = (t0) ^ (t5);\
	(t0) = ~(t4);\
	(t0) = (t1) & (t0);\
	(t4) = (t2) ^ (t0);\
	(t0) = (a2) ^ (a4);\
	(t2) = (a5) | (t3);\
	(t3) = (t1) ^ (t2);\
	(t1) = ~(t0);\
	(t1) = (t3) & (t1);\
	(t2) = (t6) ^ (t1);\
	(t1) = ~(t4);\
	(t1) = (a6) & (t1);\
	(t6) = (t1) ^ (t2);\
	(out1) = (out1) ^ (t6);\
	(t1) = ~(t2);\
	(t2) = ~(a6);\
	(t2) = (t4) & (t2);\
	(t6) = (t2) ^ (t1);\
	(out2) = (out2) ^ (t6);\
	(t2) = (t4) ^ (t1);\
	(t1) = ~(t0);\
	(t1) = (t2) & (t1);\
	(t0) = (t7) | (t1);\
	(t1) = (t3) ^ (t0);\
	(t0) = (t5) | (a6);\
	(t2) = (t0) ^ (t1);\
	(out3) = (out3) ^ (t2);\
	(t0) = (a6) & (t5);\
	(t2) = (t0) ^ (t1);\
	(out4) = (out4) ^ (t2);\

//
// s5-04832, 48 gates, 15/16 regs, 9 andn, 5/23/62/109/159 stalls, 72 biop
// Currently used for MMX/SSE2
//
#define s5(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t1) = (a1) | (a3);\
	(t10) = ~(a6);\
	(t10) = (t1) & (t10);\
	(t6) = (a1) ^ (t10);\
	(t2) = (a3) ^ (t6);\
	(t3) = (a4) | (t2);\
	(t7) = ~(a4);\
	(t7) = (t10) & (t7);\
	(t10) = (a3) ^ (t7);\
	(t7) = (a5) & (t10);\
	(t12) = (a1) | (t2);\
	(t2) = (t7) ^ (t12);\
	(t7) = (a4) ^ (t2);\
	(t2) = (a6) ^ (t7);\
	(t4) = (t6) | (t2);\
	(t8) = (a5) & (t4);\
	(t11) = (t6) ^ (t8);\
	(t9) = (a4) & (t12);\
	(t5) = (t11) ^ (t9);\
	(t11) = ~(a1);\
	(t11) = (t4) & (t11);\
	(t4) = (t10) ^ (t11);\
	(t9) = (a5) ^ (t3);\
	(t0) = ~(t4);\
	(t0) = (t9) & (t0);\
	(t4) = ~(t0);\
	(t0) = ~(a2);\
	(t0) = (t4) & (t0);\
	(t4) = (t0) ^ (t7);\
	(out3) = (out3) ^ (t4);\
	(t7) = ~(t8);\
	(t7) = (t10) & (t7);\
	(t0) = (t11) ^ (t9);\
	(t11) = (t5) | (t0);\
	(t4) = ~(t7);\
	(t4) = (t11) & (t4);\
	(t0) = ~(t4);\
	(t0) = (t3) & (t0);\
	(t11) = (t2) & (t4);\
	(t7) = (t9) ^ (t11);\
	(t2) = (t10) & (t12);\
	(t11) = (t7) | (t2);\
	(t9) = (t8) ^ (t11);\
	(t11) = (t9) & (a2);\
	(t12) = (t11) ^ (t5);\
	(out4) = (out4) ^ (t12);\
	(t12) = (t1) ^ (t4);\
	(t2) = (a1) ^ (t12);\
	(t11) = (a4) & (t7);\
	(t8) = (t2) ^ (t11);\
	(t12) = (t0) | (a2);\
	(t11) = (t12) ^ (t8);\
	(out1) = (out1) ^ (t11);\
	(t9) = (t3) ^ (t10);\
	(t5) = ~(t8);\
	(t5) = (t9) & (t5);\
	(t4) = (t6) ^ (t7);\
	(t1) = (t5) ^ (t4);\
	(t2) = (t3) & (a2);\
	(t0) = (t2) ^ (t1);\
	(out2) = (out2) ^ (t0);\

//
// s6-000007, 46 gates, 19 regs, 8 andn, 3/19/39/66/101 stalls, 69 biop
// Currently used for x86-64 SSE2
//
#define s6(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t0) = (a2) ^ (a5);\
	(t8) = (a2) | (a6);\
	(t1) = (a1) & (t8);\
	(t8) = (t0) ^ (t1);\
	(t0) = (a6) ^ (t8);\
	(t12) = ~(t0);\
	(t12) = (a5) & (t12);\
	(t11) = (a1) & (t0);\
	(t0) = (a2) ^ (t11);\
	(t4) = (a1) ^ (a3);\
	(t13) = (t0) | (t4);\
	(t2) = (t8) ^ (t13);\
	(t7) = (a3) & (t2);\
	(t6) = ~(a6);\
	(t6) = (t7) & (t6);\
	(t9) = (t12) | (t0);\
	(t0) = (t6) ^ (t9);\
	(t10) = (t0) & (a4);\
	(t5) = (t10) ^ (t2);\
	(out4) = (out4) ^ (t5);\
	(t5) = (a2) ^ (t13);\
	(t13) = ~(t5);\
	(t13) = (a6) & (t13);\
	(t10) = (a3) ^ (t13);\
	(t13) = ~(t7);\
	(t13) = (a5) & (t13);\
	(t3) = (t10) | (t13);\
	(t13) = (a1) | (t2);\
	(t2) = (t9) & (t13);\
	(t9) = (t10) ^ (t2);\
	(t13) = ~(t6);\
	(t13) = (t9) & (t13);\
	(t6) = (t12) | (a4);\
	(t12) = (t6) ^ (t13);\
	(out3) = (out3) ^ (t12);\
	(t2) = (a2) | (t4);\
	(t6) = (t0) ^ (t2);\
	(t12) = (t1) | (t3);\
	(t13) = (t6) ^ (t12);\
	(t4) = (t8) ^ (t9);\
	(t0) = ~(t4);\
	(t0) = (a5) & (t0);\
	(t1) = ~(t5);\
	(t6) = (t2) ^ (t1);\
	(t12) = (t0) ^ (t6);\
	(t9) = ~(a4);\
	(t9) = (t12) & (t9);\
	(t12) = (t9) ^ (t13);\
	(out2) = (out2) ^ (t12);\
	(t9) = (a6) ^ (t11);\
	(t8) = (a1) ^ (t10);\
	(t4) = (t9) & (t8);\
	(t5) = (t7) ^ (t6);\
	(t2) = (t4) ^ (t5);\
	(t1) = ~(a4);\
	(t1) = (t3) & (t1);\
	(t0) = (t1) ^ (t2);\
	(out1) = (out1) ^ (t0);\

//
// s7-056945, 46 gates, 16 regs, 7 andn, 10/31/62/107/156 stalls, 67 biop
// Currently used for MMX/SSE2
//
#define s7(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t6) = (a4) ^ (a5);\
	(t3) = (a3) ^ (t6);\
	(t1) = (a6) & (t3);\
	(t2) = (a4) & (t6);\
	(t4) = (a2) ^ (t2);\
	(t0) = (t1) & (t4);\
	(t7) = (a6) & (t2);\
	(t5) = (a3) ^ (t7);\
	(t7) = (t4) | (t5);\
	(t8) = (a6) ^ (t6);\
	(t6) = (t7) ^ (t8);\
	(t7) = ~(t0);\
	(t7) = (a1) & (t7);\
	(t9) = (t7) ^ (t6);\
	(out4) = (out4) ^ (t9);\
	(t7) = ~(t3);\
	(t7) = (a5) & (t7);\
	(t0) = (t4) | (t7);\
	(t9) = (t1) ^ (t5);\
	(t5) = (t0) ^ (t9);\
	(t0) = (t1) ^ (t8);\
	(t1) = ~(t0);\
	(t1) = (a4) & (t1);\
	(t0) = ~(t1);\
	(t0) = (t4) & (t0);\
	(t4) = (a5) ^ (t9);\
	(t1) = (t0) ^ (t4);\
	(t9) = (t2) | (t6);\
	(t0) = (a3) & (t1);\
	(t4) = (t9) | (t0);\
	(t2) = ~(t8);\
	(t2) = (t3) & (t2);\
	(t6) = (t4) ^ (t2);\
	(t8) = ~(a1);\
	(t8) = (t6) & (t8);\
	(t9) = (t8) ^ (t5);\
	(out1) = (out1) ^ (t9);\
	(t9) = (t1) | (t6);\
	(t8) = (a6) & (t9);\
	(t3) = (a2) & (t8);\
	(t4) = (t5) ^ (t6);\
	(t2) = (t3) ^ (t4);\
	(t9) = (t0) | (t2);\
	(t5) = (t8) ^ (t9);\
	(t3) = (a5) ^ (t4);\
	(t0) = (t5) | (t3);\
	(t9) = (t0) & (a1);\
	(t5) = (t9) ^ (t1);\
	(out3) = (out3) ^ (t5);\
	(t9) = (t8) ^ (t0);\
	(t4) = (t7) | (t9);\
	(t5) = ~(t6);\
	(t3) = (t4) ^ (t5);\
	(t1) = ~(a1);\
	(t1) = (t3) & (t1);\
	(t0) = (t1) ^ (t2);\
	(out2) = (out2) ^ (t0);\

//
// s8-004798, 41 gates, 14 regs, 7 andn, 7/35/76/118/160 stalls, 59 biop
// Currently used for MMX/SSE2
//
#define s8(a1, a2, a3, a4, a5, a6, out1, out2, out3, out4)\
	(t8) = ~(a2);\
	(t8) = (a3) & (t8);\
	(t1) = ~(a3);\
	(t1) = (a5) & (t1);\
	(t6) = (a4) ^ (t1);\
	(t1) = (a1) & (t6);\
	(t7) = ~(t8);\
	(t7) = (t1) & (t7);\
	(t3) = ~(t6);\
	(t3) = (a2) & (t3);\
	(t9) = (a1) | (t3);\
	(t0) = ~(a3);\
	(t0) = (a2) & (t0);\
	(t4) = (a5) ^ (t0);\
	(t0) = (t9) & (t4);\
	(t2) = (t1) | (t0);\
	(t1) = (t6) ^ (t0);\
	(t0) = ~(t1);\
	(t6) = ~(t9);\
	(t6) = (a3) & (t6);\
	(t1) = (t0) ^ (t6);\
	(t9) = (t8) ^ (t1);\
	(t8) = (t7) | (a6);\
	(t6) = (t8) ^ (t9);\
	(out2) = (out2) ^ (t6);\
	(t0) = (a1) ^ (t9);\
	(t6) = (a5) & (t0);\
	(t8) = (a2) ^ (t1);\
	(t9) = (t6) ^ (t8);\
	(t1) = (t3) ^ (t9);\
	(t6) = (a4) | (t8);\
	(t3) = (t1) ^ (t6);\
	(t5) = (t4) ^ (t3);\
	(t8) = (a1) ^ (t5);\
	(t6) = (t8) & (a6);\
	(t4) = (t6) ^ (t1);\
	(out4) = (out4) ^ (t4);\
	(t6) = (t2) ^ (t9);\
	(t4) = (a2) | (t6);\
	(t3) = (t0) ^ (t4);\
	(t8) = (a5) ^ (t3);\
	(t9) = (t2) & (a6);\
	(t6) = (t9) ^ (t8);\
	(out3) = (out3) ^ (t6);\
	(t9) = (a4) | (t0);\
	(t6) = ~(t9);\
	(t6) = (t8) & (t6);\
	(t4) = (t7) | (t6);\
	(t3) = (t5) ^ (t4);\
	(t2) = (t3) | (a6);\
	(t0) = (t2) ^ (t1);\
	(out1) = (out1) ^ (t0);\

#define CLEAR_BLOCK_8(i)                                                             \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 0, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 1, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 2, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 3, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 4, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 5, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 6, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 7, DES_VECTOR_ZERO); \

#define CLEAR_BLOCK()  \
	CLEAR_BLOCK_8(0);  \
	CLEAR_BLOCK_8(8);  \
	CLEAR_BLOCK_8(16); \
	CLEAR_BLOCK_8(24); \
	CLEAR_BLOCK_8(32); \
	CLEAR_BLOCK_8(40); \
	CLEAR_BLOCK_8(48); \
	CLEAR_BLOCK_8(56); \

#define w(p, q)    DES_VECTOR_XOR_FUNC(dataBlocks[p], DESContextArray[threadIdx.x].keys[q])
#define x(p)    DES_VECTOR_XOR_FUNC(dataBlocks[CUDA_expansionFunction[p]], DESContextArray[threadIdx.x].keys[keySchedule[keyScheduleIndexBase + (p)]])
#define y(p, q) DES_VECTOR_XOR_FUNC(dataBlocks[p],                         DESContextArray[threadIdx.x].keys[keySchedule[keyScheduleIndexBase + (q)]])
#define z(r)    (&dataBlocks[r])

#define EF(p) (CUDA_expansionFunction[p]<<DB_SHIFT)

DES_FUNCTION_QUALIFIERS
void DES_Crypt(volatile unsigned int keyFrom00To27, volatile unsigned int keyFrom28To48, int intSalt)
{
	DES_Vector t0;
	DES_Vector t1;
	DES_Vector t2;
	DES_Vector t3;
	DES_Vector t4;
	DES_Vector t5;
	DES_Vector t6;
	DES_Vector t7;
	DES_Vector t8;
	DES_Vector t9;
	DES_Vector t10;
	DES_Vector t11;
	DES_Vector t12;
	DES_Vector t13;

	if (threadIdx.y == 0)
		CLEAR_BLOCK();
	
	volatile DES_Vector *db = dataBlocks + threadIdx.x;
	
#define K00 ((keyFrom00To27 & (0x1U << 0)) ? 0xffffffffU : 0x0)
#define K01 ((keyFrom00To27 & (0x1U << 1)) ? 0xffffffffU : 0x0)
#define K02 ((keyFrom00To27 & (0x1U << 2)) ? 0xffffffffU : 0x0)
#define K03 ((keyFrom00To27 & (0x1U << 3)) ? 0xffffffffU : 0x0)
#define K04 ((keyFrom00To27 & (0x1U << 4)) ? 0xffffffffU : 0x0)
#define K05 ((keyFrom00To27 & (0x1U << 5)) ? 0xffffffffU : 0x0)
#define K06 ((keyFrom00To27 & (0x1U << 6)) ? 0xffffffffU : 0x0)
#define K07 ((keyFrom00To27 & (0x1U << 7)) ? 0xffffffffU : 0x0)
#define K08 ((keyFrom00To27 & (0x1U << 8)) ? 0xffffffffU : 0x0)
#define K09 ((keyFrom00To27 & (0x1U << 9)) ? 0xffffffffU : 0x0)
#define K10 ((keyFrom00To27 & (0x1U << 10)) ? 0xffffffffU : 0x0)
#define K11 ((keyFrom00To27 & (0x1U << 11)) ? 0xffffffffU : 0x0)
#define K12 ((keyFrom00To27 & (0x1U << 12)) ? 0xffffffffU : 0x0)
#define K13 ((keyFrom00To27 & (0x1U << 13)) ? 0xffffffffU : 0x0)
#define K14 ((keyFrom00To27 & (0x1U << 14)) ? 0xffffffffU : 0x0)
#define K15 ((keyFrom00To27 & (0x1U << 15)) ? 0xffffffffU : 0x0)
#define K16 ((keyFrom00To27 & (0x1U << 16)) ? 0xffffffffU : 0x0)
#define K17 ((keyFrom00To27 & (0x1U << 17)) ? 0xffffffffU : 0x0)
#define K18 ((keyFrom00To27 & (0x1U << 18)) ? 0xffffffffU : 0x0)
#define K19 ((keyFrom00To27 & (0x1U << 19)) ? 0xffffffffU : 0x0)
#define K20 ((keyFrom00To27 & (0x1U << 20)) ? 0xffffffffU : 0x0)
#define K21 ((keyFrom00To27 & (0x1U << 21)) ? 0xffffffffU : 0x0)
#define K22 ((keyFrom00To27 & (0x1U << 22)) ? 0xffffffffU : 0x0)
#define K23 ((keyFrom00To27 & (0x1U << 23)) ? 0xffffffffU : 0x0)
#define K24 ((keyFrom00To27 & (0x1U << 24)) ? 0xffffffffU : 0x0)
#define K25 ((keyFrom00To27 & (0x1U << 25)) ? 0xffffffffU : 0x0)
#define K26 ((keyFrom00To27 & (0x1U << 26)) ? 0xffffffffU : 0x0)
#define K27 ((keyFrom00To27 & (0x1U << 27)) ? 0xffffffffU : 0x0)
#define K28 ((keyFrom28To48 & (0x1U << (28 - 28))) ? 0xffffffffU : 0x0)
#define K29 ((keyFrom28To48 & (0x1U << (29 - 28))) ? 0xffffffffU : 0x0)
#define K30 ((keyFrom28To48 & (0x1U << (30 - 28))) ? 0xffffffffU : 0x0)
#define K31 ((keyFrom28To48 & (0x1U << (31 - 28))) ? 0xffffffffU : 0x0)
#define K32 ((keyFrom28To48 & (0x1U << (32 - 28))) ? 0xffffffffU : 0x0)
#define K33 ((keyFrom28To48 & (0x1U << (33 - 28))) ? 0xffffffffU : 0x0)
#define K34 ((keyFrom28To48 & (0x1U << (34 - 28))) ? 0xffffffffU : 0x0)
#define K35 ((keyFrom28To48 & (0x1U << (35 - 28))) ? 0xffffffffU : 0x0)
#define K36 ((keyFrom28To48 & (0x1U << (36 - 28))) ? 0xffffffffU : 0x0)
#define K37 ((keyFrom28To48 & (0x1U << (37 - 28))) ? 0xffffffffU : 0x0)
#define K38 ((keyFrom28To48 & (0x1U << (38 - 28))) ? 0xffffffffU : 0x0)
#define K39 ((keyFrom28To48 & (0x1U << (39 - 28))) ? 0xffffffffU : 0x0)
#define K40 ((keyFrom28To48 & (0x1U << (40 - 28))) ? 0xffffffffU : 0x0)
#define K41 ((keyFrom28To48 & (0x1U << (41 - 28))) ? 0xffffffffU : 0x0)
#define K42 ((keyFrom28To48 & (0x1U << (42 - 28))) ? 0xffffffffU : 0x0)
#define K43 ((keyFrom28To48 & (0x1U << (43 - 28))) ? 0xffffffffU : 0x0)
#define K44 ((keyFrom28To48 & (0x1U << (44 - 28))) ? 0xffffffffU : 0x0)
#define K45 ((keyFrom28To48 & (0x1U << (45 - 28))) ? 0xffffffffU : 0x0)
#define K46 ((keyFrom28To48 & (0x1U << (46 - 28))) ? 0xffffffffU : 0x0)
#define K47 ((keyFrom28To48 & (0x1U << (47 - 28))) ? 0xffffffffU : 0x0)
#define K48 ((keyFrom28To48 & (0x1U << (48 - 28))) ? 0xffffffffU : 0x0)

#define K00XOR(val) ((keyFrom00To27 & (0x1U << 0)) ? ~(val) : (val))
#define K01XOR(val) ((keyFrom00To27 & (0x1U << 1)) ? ~(val) : (val))
#define K02XOR(val) ((keyFrom00To27 & (0x1U << 2)) ? ~(val) : (val))
#define K03XOR(val) ((keyFrom00To27 & (0x1U << 3)) ? ~(val) : (val))
#define K04XOR(val) ((keyFrom00To27 & (0x1U << 4)) ? ~(val) : (val))
#define K05XOR(val) ((keyFrom00To27 & (0x1U << 5)) ? ~(val) : (val))
#define K06XOR(val) ((keyFrom00To27 & (0x1U << 6)) ? ~(val) : (val))
#define K07XOR(val) ((keyFrom00To27 & (0x1U << 7)) ? ~(val) : (val))
#define K08XOR(val) ((keyFrom00To27 & (0x1U << 8)) ? ~(val) : (val))
#define K09XOR(val) ((keyFrom00To27 & (0x1U << 9)) ? ~(val) : (val))
#define K10XOR(val) ((keyFrom00To27 & (0x1U << 10)) ? ~(val) : (val))
#define K11XOR(val) ((keyFrom00To27 & (0x1U << 11)) ? ~(val) : (val))
#define K12XOR(val) ((keyFrom00To27 & (0x1U << 12)) ? ~(val) : (val))
#define K13XOR(val) ((keyFrom00To27 & (0x1U << 13)) ? ~(val) : (val))
#define K14XOR(val) ((keyFrom00To27 & (0x1U << 14)) ? ~(val) : (val))
#define K15XOR(val) ((keyFrom00To27 & (0x1U << 15)) ? ~(val) : (val))
#define K16XOR(val) ((keyFrom00To27 & (0x1U << 16)) ? ~(val) : (val))
#define K17XOR(val) ((keyFrom00To27 & (0x1U << 17)) ? ~(val) : (val))
#define K18XOR(val) ((keyFrom00To27 & (0x1U << 18)) ? ~(val) : (val))
#define K19XOR(val) ((keyFrom00To27 & (0x1U << 19)) ? ~(val) : (val))
#define K20XOR(val) ((keyFrom00To27 & (0x1U << 20)) ? ~(val) : (val))
#define K21XOR(val) ((keyFrom00To27 & (0x1U << 21)) ? ~(val) : (val))
#define K22XOR(val) ((keyFrom00To27 & (0x1U << 22)) ? ~(val) : (val))
#define K23XOR(val) ((keyFrom00To27 & (0x1U << 23)) ? ~(val) : (val))
#define K24XOR(val) ((keyFrom00To27 & (0x1U << 24)) ? ~(val) : (val))
#define K25XOR(val) ((keyFrom00To27 & (0x1U << 25)) ? ~(val) : (val))
#define K26XOR(val) ((keyFrom00To27 & (0x1U << 26)) ? ~(val) : (val))
#define K27XOR(val) ((keyFrom00To27 & (0x1U << 27)) ? ~(val) : (val))
#define K28XOR(val) ((keyFrom28To48 & (0x1U << (28 - 28))) ? ~(val) : (val))
#define K29XOR(val) ((keyFrom28To48 & (0x1U << (29 - 28))) ? ~(val) : (val))
#define K30XOR(val) ((keyFrom28To48 & (0x1U << (30 - 28))) ? ~(val) : (val))
#define K31XOR(val) ((keyFrom28To48 & (0x1U << (31 - 28))) ? ~(val) : (val))
#define K32XOR(val) ((keyFrom28To48 & (0x1U << (32 - 28))) ? ~(val) : (val))
#define K33XOR(val) ((keyFrom28To48 & (0x1U << (33 - 28))) ? ~(val) : (val))
#define K34XOR(val) ((keyFrom28To48 & (0x1U << (34 - 28))) ? ~(val) : (val))
#define K35XOR(val) ((keyFrom28To48 & (0x1U << (35 - 28))) ? ~(val) : (val))
#define K36XOR(val) ((keyFrom28To48 & (0x1U << (36 - 28))) ? ~(val) : (val))
#define K37XOR(val) ((keyFrom28To48 & (0x1U << (37 - 28))) ? ~(val) : (val))
#define K38XOR(val) ((keyFrom28To48 & (0x1U << (38 - 28))) ? ~(val) : (val))
#define K39XOR(val) ((keyFrom28To48 & (0x1U << (39 - 28))) ? ~(val) : (val))
#define K40XOR(val) ((keyFrom28To48 & (0x1U << (40 - 28))) ? ~(val) : (val))
#define K41XOR(val) ((keyFrom28To48 & (0x1U << (41 - 28))) ? ~(val) : (val))
#define K42XOR(val) ((keyFrom28To48 & (0x1U << (42 - 28))) ? ~(val) : (val))
#define K43XOR(val) ((keyFrom28To48 & (0x1U << (43 - 28))) ? ~(val) : (val))
#define K44XOR(val) ((keyFrom28To48 & (0x1U << (44 - 28))) ? ~(val) : (val))
#define K45XOR(val) ((keyFrom28To48 & (0x1U << (45 - 28))) ? ~(val) : (val))
#define K46XOR(val) ((keyFrom28To48 & (0x1U << (46 - 28))) ? ~(val) : (val))
#define K47XOR(val) ((keyFrom28To48 & (0x1U << (47 - 28))) ? ~(val) : (val))
#define K48XOR(val) ((keyFrom28To48 & (0x1U << (48 - 28))) ? ~(val) : (val))
#define K49XOR(val) ((val) ^ CUDA_keyFrom49To55Array[0])
#define K50XOR(val) ((val) ^ CUDA_keyFrom49To55Array[1])
#define K51XOR(val) ((val) ^ CUDA_keyFrom49To55Array[2])
#define K52XOR(val) ((val) ^ CUDA_keyFrom49To55Array[3])
#define K53XOR(val) ((val) ^ CUDA_keyFrom49To55Array[4])
#define K54XOR(val) ((val) ^ CUDA_keyFrom49To55Array[5])
#define K55XOR(val) ((val) ^ CUDA_keyFrom49To55Array[6])

#define SALT 0 // intSalt

	volatile DES_Vector DB00 = db[ 0<<DB_SHIFT];
	volatile DES_Vector DB01 = db[ 1<<DB_SHIFT];
	volatile DES_Vector DB02 = db[ 2<<DB_SHIFT];
	volatile DES_Vector DB03 = 0;
	volatile DES_Vector DB04 = 0;
	volatile DES_Vector DB05 = 0;
	volatile DES_Vector DB06 = 0;
	volatile DES_Vector DB07 = 0;
	volatile DES_Vector DB08 = 0;
	volatile DES_Vector DB09 = 0;
	volatile DES_Vector DB10 = 0;
	volatile DES_Vector DB11 = 0;
	volatile DES_Vector DB12 = 0;
	volatile DES_Vector DB13 = 0;
	volatile DES_Vector DB14 = 0;
	volatile DES_Vector DB15 = db[15<<DB_SHIFT];
	volatile DES_Vector DB16 = db[16<<DB_SHIFT];
	volatile DES_Vector DB17 = db[17<<DB_SHIFT];
	volatile DES_Vector DB18 = db[18<<DB_SHIFT];
	volatile DES_Vector DB19 = 0;
	volatile DES_Vector DB20 = 0;
	volatile DES_Vector DB21 = 0;
	volatile DES_Vector DB22 = 0;
	volatile DES_Vector DB23 = 0;
	volatile DES_Vector DB24 = 0;
	volatile DES_Vector DB25 = 0;
	volatile DES_Vector DB26 = 0;
	volatile DES_Vector DB27 = 0;
	volatile DES_Vector DB28 = 0;
	volatile DES_Vector DB29 = 0;
	volatile DES_Vector DB30 = 0;
	volatile DES_Vector DB31 = db[31<<DB_SHIFT];
	for (int i = 0; i < 13; ++i) {
		// ROUND_A(0);
		if (!threadIdx.y) {
		s1(K12XOR((   1 & SALT) ? DB15 : DB31), K46XOR((   2 & SALT) ? DB16 : DB00), K33XOR((   4 & SALT) ? DB17 : DB01), K52XOR((   8 & SALT) ? DB18 : DB02), K48XOR((  16 & SALT) ? DB19 : DB03), K20XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K34XOR((  64 & SALT) ? DB19 : DB03), K55XOR(( 128 & SALT) ? DB20 : DB04), K05XOR(( 256 & SALT) ? DB21 : DB05), K13XOR(( 512 & SALT) ? DB22 : DB06), K18XOR((1024 & SALT) ? DB23 : DB07), K40XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K04XOR(                DB07       ), K32XOR(                DB08       ), K26XOR(                DB09       ), K27XOR(                DB10       ), K38XOR(                DB11       ), K54XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K53XOR(                DB11       ), K06XOR(                DB12       ), K31XOR(                DB13       ), K25XOR(                DB14       ), K19XOR(                DB15       ), K41XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K15XOR((   1 & SALT) ? DB31 : DB15), K24XOR((   2 & SALT) ? DB00 : DB16), K28XOR((   4 & SALT) ? DB01 : DB17), K43XOR((   8 & SALT) ? DB02 : DB18), K30XOR((  16 & SALT) ? DB03 : DB19), K03XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K35XOR((  64 & SALT) ? DB03 : DB19), K22XOR(( 128 & SALT) ? DB04 : DB20), K02XOR(( 256 & SALT) ? DB05 : DB21), K44XOR(( 512 & SALT) ? DB06 : DB22), K14XOR((1024 & SALT) ? DB07 : DB23), K23XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K51XOR(                DB23       ), K16XOR(                DB24       ), K29XOR(                DB25       ), K49XOR(                DB26       ), K07XOR(                DB27       ), K17XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K37XOR(                DB27       ), K08XOR(                DB28       ), K09XOR(                DB29       ), K50XOR(                DB30       ), K42XOR(                DB31       ), K21XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(0);
		if (!threadIdx.y) {
		s1(K05XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K39XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K26XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K45XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K41XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K13XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K27XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K48XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K53XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K06XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K11XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K33XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K52XOR(db[39<<DB_SHIFT]), K25XOR(db[40<<DB_SHIFT]), K19XOR(db[41<<DB_SHIFT]), K20XOR(db[42<<DB_SHIFT]), K31XOR(db[43<<DB_SHIFT]), K47XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K46XOR(db[43<<DB_SHIFT]), K54XOR(db[44<<DB_SHIFT]), K55XOR(db[45<<DB_SHIFT]), K18XOR(db[46<<DB_SHIFT]), K12XOR(db[47<<DB_SHIFT]), K34XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K08XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K17XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K21XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K36XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K23XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K49XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K28XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K15XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K24XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K37XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K07XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K16XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K44XOR(db[55<<DB_SHIFT]), K09XOR(db[56<<DB_SHIFT]), K22XOR(db[57<<DB_SHIFT]), K42XOR(db[58<<DB_SHIFT]), K00XOR(db[59<<DB_SHIFT]), K10XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K30XOR(db[59<<DB_SHIFT]), K01XOR(db[60<<DB_SHIFT]), K02XOR(db[61<<DB_SHIFT]), K43XOR(db[62<<DB_SHIFT]), K35XOR(db[63<<DB_SHIFT]), K14XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(96);
		if (!threadIdx.y) {
		s1(K46XOR((   1 & SALT) ? DB15 : DB31), K25XOR((   2 & SALT) ? DB16 : DB00), K12XOR((   4 & SALT) ? DB17 : DB01), K31XOR((   8 & SALT) ? DB18 : DB02), K27XOR((  16 & SALT) ? DB19 : DB03), K54XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K13XOR((  64 & SALT) ? DB19 : DB03), K34XOR(( 128 & SALT) ? DB20 : DB04), K39XOR(( 256 & SALT) ? DB21 : DB05), K47XOR(( 512 & SALT) ? DB22 : DB06), K52XOR((1024 & SALT) ? DB23 : DB07), K19XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K38XOR(                DB07       ), K11XOR(                DB08       ), K05XOR(                DB09       ), K06XOR(                DB10       ), K48XOR(                DB11       ), K33XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K32XOR(                DB11       ), K40XOR(                DB12       ), K41XOR(                DB13       ), K04XOR(                DB14       ), K53XOR(                DB15       ), K20XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K51XOR((   1 & SALT) ? DB31 : DB15), K03XOR((   2 & SALT) ? DB00 : DB16), K07XOR((   4 & SALT) ? DB01 : DB17), K22XOR((   8 & SALT) ? DB02 : DB18), K09XOR((  16 & SALT) ? DB03 : DB19), K35XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K14XOR((  64 & SALT) ? DB03 : DB19), K01XOR(( 128 & SALT) ? DB04 : DB20), K10XOR(( 256 & SALT) ? DB05 : DB21), K23XOR(( 512 & SALT) ? DB06 : DB22), K50XOR((1024 & SALT) ? DB07 : DB23), K02XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K30XOR(                DB23       ), K24XOR(                DB24       ), K08XOR(                DB25       ), K28XOR(                DB26       ), K43XOR(                DB27       ), K49XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K16XOR(                DB27       ), K44XOR(                DB28       ), K17XOR(                DB29       ), K29XOR(                DB30       ), K21XOR(                DB31       ), K00XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(96);
		if (!threadIdx.y) {
		s1(K32XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K11XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K53XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K48XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K13XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K40XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K54XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K20XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K25XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K33XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K38XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K05XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K55XOR(db[39<<DB_SHIFT]), K52XOR(db[40<<DB_SHIFT]), K46XOR(db[41<<DB_SHIFT]), K47XOR(db[42<<DB_SHIFT]), K34XOR(db[43<<DB_SHIFT]), K19XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K18XOR(db[43<<DB_SHIFT]), K26XOR(db[44<<DB_SHIFT]), K27XOR(db[45<<DB_SHIFT]), K45XOR(db[46<<DB_SHIFT]), K39XOR(db[47<<DB_SHIFT]), K06XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K37XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K42XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K50XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K08XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K24XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K21XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K00XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K44XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K49XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K09XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K36XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K17XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K16XOR(db[55<<DB_SHIFT]), K10XOR(db[56<<DB_SHIFT]), K51XOR(db[57<<DB_SHIFT]), K14XOR(db[58<<DB_SHIFT]), K29XOR(db[59<<DB_SHIFT]), K35XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K02XOR(db[59<<DB_SHIFT]), K30XOR(db[60<<DB_SHIFT]), K03XOR(db[61<<DB_SHIFT]), K15XOR(db[62<<DB_SHIFT]), K07XOR(db[63<<DB_SHIFT]), K43XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(192);
		if (!threadIdx.y) {
		s1(K18XOR((   1 & SALT) ? DB15 : DB31), K52XOR((   2 & SALT) ? DB16 : DB00), K39XOR((   4 & SALT) ? DB17 : DB01), K34XOR((   8 & SALT) ? DB18 : DB02), K54XOR((  16 & SALT) ? DB19 : DB03), K26XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K40XOR((  64 & SALT) ? DB19 : DB03), K06XOR(( 128 & SALT) ? DB20 : DB04), K11XOR(( 256 & SALT) ? DB21 : DB05), K19XOR(( 512 & SALT) ? DB22 : DB06), K55XOR((1024 & SALT) ? DB23 : DB07), K46XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K41XOR(                DB07       ), K38XOR(                DB08       ), K32XOR(                DB09       ), K33XOR(                DB10       ), K20XOR(                DB11       ), K05XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K04XOR(                DB11       ), K12XOR(                DB12       ), K13XOR(                DB13       ), K31XOR(                DB14       ), K25XOR(                DB15       ), K47XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K23XOR((   1 & SALT) ? DB31 : DB15), K28XOR((   2 & SALT) ? DB00 : DB16), K36XOR((   4 & SALT) ? DB01 : DB17), K51XOR((   8 & SALT) ? DB02 : DB18), K10XOR((  16 & SALT) ? DB03 : DB19), K07XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K43XOR((  64 & SALT) ? DB03 : DB19), K30XOR(( 128 & SALT) ? DB04 : DB20), K35XOR(( 256 & SALT) ? DB05 : DB21), K24XOR(( 512 & SALT) ? DB06 : DB22), K22XOR((1024 & SALT) ? DB07 : DB23), K03XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K02XOR(                DB23       ), K49XOR(                DB24       ), K37XOR(                DB25       ), K00XOR(                DB26       ), K15XOR(                DB27       ), K21XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K17XOR(                DB27       ), K16XOR(                DB28       ), K42XOR(                DB29       ), K01XOR(                DB30       ), K50XOR(                DB31       ), K29XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(192);
		if (!threadIdx.y) {
		s1(K04XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K38XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K25XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K20XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K40XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K12XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K26XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K47XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K52XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K05XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K41XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K32XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K27XOR(db[39<<DB_SHIFT]), K55XOR(db[40<<DB_SHIFT]), K18XOR(db[41<<DB_SHIFT]), K19XOR(db[42<<DB_SHIFT]), K06XOR(db[43<<DB_SHIFT]), K46XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K45XOR(db[43<<DB_SHIFT]), K53XOR(db[44<<DB_SHIFT]), K54XOR(db[45<<DB_SHIFT]), K48XOR(db[46<<DB_SHIFT]), K11XOR(db[47<<DB_SHIFT]), K33XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K09XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K14XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K22XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K37XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K49XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K50XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K29XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K16XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K21XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K10XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K08XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K42XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K17XOR(db[55<<DB_SHIFT]), K35XOR(db[56<<DB_SHIFT]), K23XOR(db[57<<DB_SHIFT]), K43XOR(db[58<<DB_SHIFT]), K01XOR(db[59<<DB_SHIFT]), K07XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K03XOR(db[59<<DB_SHIFT]), K02XOR(db[60<<DB_SHIFT]), K28XOR(db[61<<DB_SHIFT]), K44XOR(db[62<<DB_SHIFT]), K36XOR(db[63<<DB_SHIFT]), K15XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(288);
		if (!threadIdx.y) {
		s1(K45XOR((   1 & SALT) ? DB15 : DB31), K55XOR((   2 & SALT) ? DB16 : DB00), K11XOR((   4 & SALT) ? DB17 : DB01), K06XOR((   8 & SALT) ? DB18 : DB02), K26XOR((  16 & SALT) ? DB19 : DB03), K53XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K12XOR((  64 & SALT) ? DB19 : DB03), K33XOR(( 128 & SALT) ? DB20 : DB04), K38XOR(( 256 & SALT) ? DB21 : DB05), K46XOR(( 512 & SALT) ? DB22 : DB06), K27XOR((1024 & SALT) ? DB23 : DB07), K18XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K13XOR(                DB07       ), K41XOR(                DB08       ), K04XOR(                DB09       ), K05XOR(                DB10       ), K47XOR(                DB11       ), K32XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K31XOR(                DB11       ), K39XOR(                DB12       ), K40XOR(                DB13       ), K34XOR(                DB14       ), K52XOR(                DB15       ), K19XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K24XOR((   1 & SALT) ? DB31 : DB15), K00XOR((   2 & SALT) ? DB00 : DB16), K08XOR((   4 & SALT) ? DB01 : DB17), K23XOR((   8 & SALT) ? DB02 : DB18), K35XOR((  16 & SALT) ? DB03 : DB19), K36XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K15XOR((  64 & SALT) ? DB03 : DB19), K02XOR(( 128 & SALT) ? DB04 : DB20), K07XOR(( 256 & SALT) ? DB05 : DB21), K49XOR(( 512 & SALT) ? DB06 : DB22), K51XOR((1024 & SALT) ? DB07 : DB23), K28XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K03XOR(                DB23       ), K21XOR(                DB24       ), K09XOR(                DB25       ), K29XOR(                DB26       ), K44XOR(                DB27       ), K50XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K42XOR(                DB27       ), K17XOR(                DB28       ), K14XOR(                DB29       ), K30XOR(                DB30       ), K22XOR(                DB31       ), K01XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(288);
		if (!threadIdx.y) {
		s1(K31XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K41XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K52XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K47XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K12XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K39XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K53XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K19XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K55XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K32XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K13XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K04XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K54XOR(db[39<<DB_SHIFT]), K27XOR(db[40<<DB_SHIFT]), K45XOR(db[41<<DB_SHIFT]), K46XOR(db[42<<DB_SHIFT]), K33XOR(db[43<<DB_SHIFT]), K18XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K48XOR(db[43<<DB_SHIFT]), K25XOR(db[44<<DB_SHIFT]), K26XOR(db[45<<DB_SHIFT]), K20XOR(db[46<<DB_SHIFT]), K38XOR(db[47<<DB_SHIFT]), K05XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K10XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K43XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K51XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K09XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K21XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K22XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K01XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K17XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K50XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K35XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K37XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K14XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K42XOR(db[55<<DB_SHIFT]), K07XOR(db[56<<DB_SHIFT]), K24XOR(db[57<<DB_SHIFT]), K15XOR(db[58<<DB_SHIFT]), K30XOR(db[59<<DB_SHIFT]), K36XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K28XOR(db[59<<DB_SHIFT]), K03XOR(db[60<<DB_SHIFT]), K00XOR(db[61<<DB_SHIFT]), K16XOR(db[62<<DB_SHIFT]), K08XOR(db[63<<DB_SHIFT]), K44XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(384);
		if (!threadIdx.y) {
		s1(K55XOR((   1 & SALT) ? DB15 : DB31), K34XOR((   2 & SALT) ? DB16 : DB00), K45XOR((   4 & SALT) ? DB17 : DB01), K40XOR((   8 & SALT) ? DB18 : DB02), K05XOR((  16 & SALT) ? DB19 : DB03), K32XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K46XOR((  64 & SALT) ? DB19 : DB03), K12XOR(( 128 & SALT) ? DB20 : DB04), K48XOR(( 256 & SALT) ? DB21 : DB05), K25XOR(( 512 & SALT) ? DB22 : DB06), K06XOR((1024 & SALT) ? DB23 : DB07), K52XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K47XOR(                DB07       ), K20XOR(                DB08       ), K38XOR(                DB09       ), K39XOR(                DB10       ), K26XOR(                DB11       ), K11XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K41XOR(                DB11       ), K18XOR(                DB12       ), K19XOR(                DB13       ), K13XOR(                DB14       ), K31XOR(                DB15       ), K53XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K03XOR((   1 & SALT) ? DB31 : DB15), K36XOR((   2 & SALT) ? DB00 : DB16), K44XOR((   4 & SALT) ? DB01 : DB17), K02XOR((   8 & SALT) ? DB02 : DB18), K14XOR((  16 & SALT) ? DB03 : DB19), K15XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K51XOR((  64 & SALT) ? DB03 : DB19), K10XOR(( 128 & SALT) ? DB04 : DB20), K43XOR(( 256 & SALT) ? DB05 : DB21), K28XOR(( 512 & SALT) ? DB06 : DB22), K30XOR((1024 & SALT) ? DB07 : DB23), K07XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K35XOR(                DB23       ), K00XOR(                DB24       ), K17XOR(                DB25       ), K08XOR(                DB26       ), K23XOR(                DB27       ), K29XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K21XOR(                DB27       ), K49XOR(                DB28       ), K50XOR(                DB29       ), K09XOR(                DB30       ), K01XOR(                DB31       ), K37XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(384);
		if (!threadIdx.y) {
		s1(K41XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K20XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K31XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K26XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K46XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K18XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K32XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K53XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K34XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K11XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K47XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K38XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K33XOR(db[39<<DB_SHIFT]), K06XOR(db[40<<DB_SHIFT]), K55XOR(db[41<<DB_SHIFT]), K25XOR(db[42<<DB_SHIFT]), K12XOR(db[43<<DB_SHIFT]), K52XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K27XOR(db[43<<DB_SHIFT]), K04XOR(db[44<<DB_SHIFT]), K05XOR(db[45<<DB_SHIFT]), K54XOR(db[46<<DB_SHIFT]), K48XOR(db[47<<DB_SHIFT]), K39XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K42XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K22XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K30XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K17XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K00XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K01XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K37XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K49XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K29XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K14XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K16XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K50XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K21XOR(db[55<<DB_SHIFT]), K43XOR(db[56<<DB_SHIFT]), K03XOR(db[57<<DB_SHIFT]), K51XOR(db[58<<DB_SHIFT]), K09XOR(db[59<<DB_SHIFT]), K15XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K07XOR(db[59<<DB_SHIFT]), K35XOR(db[60<<DB_SHIFT]), K36XOR(db[61<<DB_SHIFT]), K24XOR(db[62<<DB_SHIFT]), K44XOR(db[63<<DB_SHIFT]), K23XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(480);
		if (!threadIdx.y) {
		s1(K27XOR((   1 & SALT) ? DB15 : DB31), K06XOR((   2 & SALT) ? DB16 : DB00), K48XOR((   4 & SALT) ? DB17 : DB01), K12XOR((   8 & SALT) ? DB18 : DB02), K32XOR((  16 & SALT) ? DB19 : DB03), K04XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K18XOR((  64 & SALT) ? DB19 : DB03), K39XOR(( 128 & SALT) ? DB20 : DB04), K20XOR(( 256 & SALT) ? DB21 : DB05), K52XOR(( 512 & SALT) ? DB22 : DB06), K33XOR((1024 & SALT) ? DB23 : DB07), K55XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K19XOR(                DB07       ), K47XOR(                DB08       ), K41XOR(                DB09       ), K11XOR(                DB10       ), K53XOR(                DB11       ), K38XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K13XOR(                DB11       ), K45XOR(                DB12       ), K46XOR(                DB13       ), K40XOR(                DB14       ), K34XOR(                DB15       ), K25XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K28XOR((   1 & SALT) ? DB31 : DB15), K08XOR((   2 & SALT) ? DB00 : DB16), K16XOR((   4 & SALT) ? DB01 : DB17), K03XOR((   8 & SALT) ? DB02 : DB18), K43XOR((  16 & SALT) ? DB03 : DB19), K44XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K23XOR((  64 & SALT) ? DB03 : DB19), K35XOR(( 128 & SALT) ? DB04 : DB20), K15XOR(( 256 & SALT) ? DB05 : DB21), K00XOR(( 512 & SALT) ? DB06 : DB22), K02XOR((1024 & SALT) ? DB07 : DB23), K36XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K07XOR(                DB23       ), K29XOR(                DB24       ), K42XOR(                DB25       ), K37XOR(                DB26       ), K24XOR(                DB27       ), K01XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K50XOR(                DB27       ), K21XOR(                DB28       ), K22XOR(                DB29       ), K10XOR(                DB30       ), K30XOR(                DB31       ), K09XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(480);
		if (!threadIdx.y) {
		s1(K13XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K47XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K34XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K53XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K18XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K45XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K04XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K25XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K06XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K38XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K19XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K41XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K05XOR(db[39<<DB_SHIFT]), K33XOR(db[40<<DB_SHIFT]), K27XOR(db[41<<DB_SHIFT]), K52XOR(db[42<<DB_SHIFT]), K39XOR(db[43<<DB_SHIFT]), K55XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K54XOR(db[43<<DB_SHIFT]), K31XOR(db[44<<DB_SHIFT]), K32XOR(db[45<<DB_SHIFT]), K26XOR(db[46<<DB_SHIFT]), K20XOR(db[47<<DB_SHIFT]), K11XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K14XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K51XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K02XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K42XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K29XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K30XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K09XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K21XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K01XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K43XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K17XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K22XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K50XOR(db[55<<DB_SHIFT]), K15XOR(db[56<<DB_SHIFT]), K28XOR(db[57<<DB_SHIFT]), K23XOR(db[58<<DB_SHIFT]), K10XOR(db[59<<DB_SHIFT]), K44XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K36XOR(db[59<<DB_SHIFT]), K07XOR(db[60<<DB_SHIFT]), K08XOR(db[61<<DB_SHIFT]), K49XOR(db[62<<DB_SHIFT]), K16XOR(db[63<<DB_SHIFT]), K24XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(576);
		if (!threadIdx.y) {
		s1(K54XOR((   1 & SALT) ? DB15 : DB31), K33XOR((   2 & SALT) ? DB16 : DB00), K20XOR((   4 & SALT) ? DB17 : DB01), K39XOR((   8 & SALT) ? DB18 : DB02), K04XOR((  16 & SALT) ? DB19 : DB03), K31XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K45XOR((  64 & SALT) ? DB19 : DB03), K11XOR(( 128 & SALT) ? DB20 : DB04), K47XOR(( 256 & SALT) ? DB21 : DB05), K55XOR(( 512 & SALT) ? DB22 : DB06), K05XOR((1024 & SALT) ? DB23 : DB07), K27XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K46XOR(                DB07       ), K19XOR(                DB08       ), K13XOR(                DB09       ), K38XOR(                DB10       ), K25XOR(                DB11       ), K41XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K40XOR(                DB11       ), K48XOR(                DB12       ), K18XOR(                DB13       ), K12XOR(                DB14       ), K06XOR(                DB15       ), K52XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K00XOR((   1 & SALT) ? DB31 : DB15), K37XOR((   2 & SALT) ? DB00 : DB16), K17XOR((   4 & SALT) ? DB01 : DB17), K28XOR((   8 & SALT) ? DB02 : DB18), K15XOR((  16 & SALT) ? DB03 : DB19), K16XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K24XOR((  64 & SALT) ? DB03 : DB19), K07XOR(( 128 & SALT) ? DB04 : DB20), K44XOR(( 256 & SALT) ? DB05 : DB21), K29XOR(( 512 & SALT) ? DB06 : DB22), K03XOR((1024 & SALT) ? DB07 : DB23), K08XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K36XOR(                DB23       ), K01XOR(                DB24       ), K14XOR(                DB25       ), K09XOR(                DB26       ), K49XOR(                DB27       ), K30XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K22XOR(                DB27       ), K50XOR(                DB28       ), K51XOR(                DB29       ), K35XOR(                DB30       ), K02XOR(                DB31       ), K10XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(576);
		if (!threadIdx.y) {
		s1(K40XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K19XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K06XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K25XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K45XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K48XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K31XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K52XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K33XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K41XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K46XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K13XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K32XOR(db[39<<DB_SHIFT]), K05XOR(db[40<<DB_SHIFT]), K54XOR(db[41<<DB_SHIFT]), K55XOR(db[42<<DB_SHIFT]), K11XOR(db[43<<DB_SHIFT]), K27XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K26XOR(db[43<<DB_SHIFT]), K34XOR(db[44<<DB_SHIFT]), K04XOR(db[45<<DB_SHIFT]), K53XOR(db[46<<DB_SHIFT]), K47XOR(db[47<<DB_SHIFT]), K38XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K43XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K23XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K03XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K14XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K01XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K02XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K10XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K50XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K30XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K15XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K42XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K51XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K22XOR(db[55<<DB_SHIFT]), K44XOR(db[56<<DB_SHIFT]), K00XOR(db[57<<DB_SHIFT]), K24XOR(db[58<<DB_SHIFT]), K35XOR(db[59<<DB_SHIFT]), K16XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K08XOR(db[59<<DB_SHIFT]), K36XOR(db[60<<DB_SHIFT]), K37XOR(db[61<<DB_SHIFT]), K21XOR(db[62<<DB_SHIFT]), K17XOR(db[63<<DB_SHIFT]), K49XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(672);
		if (!threadIdx.y) {
		s1(K26XOR((   1 & SALT) ? DB15 : DB31), K05XOR((   2 & SALT) ? DB16 : DB00), K47XOR((   4 & SALT) ? DB17 : DB01), K11XOR((   8 & SALT) ? DB18 : DB02), K31XOR((  16 & SALT) ? DB19 : DB03), K34XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K48XOR((  64 & SALT) ? DB19 : DB03), K38XOR(( 128 & SALT) ? DB20 : DB04), K19XOR(( 256 & SALT) ? DB21 : DB05), K27XOR(( 512 & SALT) ? DB22 : DB06), K32XOR((1024 & SALT) ? DB23 : DB07), K54XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K18XOR(                DB07       ), K46XOR(                DB08       ), K40XOR(                DB09       ), K41XOR(                DB10       ), K52XOR(                DB11       ), K13XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K12XOR(                DB11       ), K20XOR(                DB12       ), K45XOR(                DB13       ), K39XOR(                DB14       ), K33XOR(                DB15       ), K55XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K29XOR((   1 & SALT) ? DB31 : DB15), K09XOR((   2 & SALT) ? DB00 : DB16), K42XOR((   4 & SALT) ? DB01 : DB17), K00XOR((   8 & SALT) ? DB02 : DB18), K44XOR((  16 & SALT) ? DB03 : DB19), K17XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K49XOR((  64 & SALT) ? DB03 : DB19), K36XOR(( 128 & SALT) ? DB04 : DB20), K16XOR(( 256 & SALT) ? DB05 : DB21), K01XOR(( 512 & SALT) ? DB06 : DB22), K28XOR((1024 & SALT) ? DB07 : DB23), K37XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K08XOR(                DB23       ), K30XOR(                DB24       ), K43XOR(                DB25       ), K10XOR(                DB26       ), K21XOR(                DB27       ), K02XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K51XOR(                DB27       ), K22XOR(                DB28       ), K23XOR(                DB29       ), K07XOR(                DB30       ), K03XOR(                DB31       ), K35XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(672);
		if (!threadIdx.y) {
		s1(K19XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K53XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K40XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K04XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K55XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K27XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K41XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K31XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K12XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K20XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K25XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K47XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K11XOR(db[39<<DB_SHIFT]), K39XOR(db[40<<DB_SHIFT]), K33XOR(db[41<<DB_SHIFT]), K34XOR(db[42<<DB_SHIFT]), K45XOR(db[43<<DB_SHIFT]), K06XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K05XOR(db[43<<DB_SHIFT]), K13XOR(db[44<<DB_SHIFT]), K38XOR(db[45<<DB_SHIFT]), K32XOR(db[46<<DB_SHIFT]), K26XOR(db[47<<DB_SHIFT]), K48XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K22XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K02XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K35XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K50XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K37XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K10XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K42XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K29XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K09XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K51XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K21XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K30XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K01XOR(db[55<<DB_SHIFT]), K23XOR(db[56<<DB_SHIFT]), K36XOR(db[57<<DB_SHIFT]), K03XOR(db[58<<DB_SHIFT]), K14XOR(db[59<<DB_SHIFT]), K24XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K44XOR(db[59<<DB_SHIFT]), K15XOR(db[60<<DB_SHIFT]), K16XOR(db[61<<DB_SHIFT]), K00XOR(db[62<<DB_SHIFT]), K49XOR(db[63<<DB_SHIFT]), K28XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		if (i >= 12)
			break;

		// ROUND_B(-48);
		if (!threadIdx.y) {
		s1(K12XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K46XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K33XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K52XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K48XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K20XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K34XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K55XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K05XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K13XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K18XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K40XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K04XOR(db[39<<DB_SHIFT]), K32XOR(db[40<<DB_SHIFT]), K26XOR(db[41<<DB_SHIFT]), K27XOR(db[42<<DB_SHIFT]), K38XOR(db[43<<DB_SHIFT]), K54XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K53XOR(db[43<<DB_SHIFT]), K06XOR(db[44<<DB_SHIFT]), K31XOR(db[45<<DB_SHIFT]), K25XOR(db[46<<DB_SHIFT]), K19XOR(db[47<<DB_SHIFT]), K41XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K15XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K24XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K28XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K43XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K30XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K03XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K35XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K22XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K02XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K44XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K14XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K23XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K51XOR(db[55<<DB_SHIFT]), K16XOR(db[56<<DB_SHIFT]), K29XOR(db[57<<DB_SHIFT]), K49XOR(db[58<<DB_SHIFT]), K07XOR(db[59<<DB_SHIFT]), K17XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K37XOR(db[59<<DB_SHIFT]), K08XOR(db[60<<DB_SHIFT]), K09XOR(db[61<<DB_SHIFT]), K50XOR(db[62<<DB_SHIFT]), K42XOR(db[63<<DB_SHIFT]), K21XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(48);
		if (!threadIdx.y) {
		s1(K05XOR((   1 & SALT) ? DB15 : DB31), K39XOR((   2 & SALT) ? DB16 : DB00), K26XOR((   4 & SALT) ? DB17 : DB01), K45XOR((   8 & SALT) ? DB18 : DB02), K41XOR((  16 & SALT) ? DB19 : DB03), K13XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K27XOR((  64 & SALT) ? DB19 : DB03), K48XOR(( 128 & SALT) ? DB20 : DB04), K53XOR(( 256 & SALT) ? DB21 : DB05), K06XOR(( 512 & SALT) ? DB22 : DB06), K11XOR((1024 & SALT) ? DB23 : DB07), K33XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K52XOR(                DB07       ), K25XOR(                DB08       ), K19XOR(                DB09       ), K20XOR(                DB10       ), K31XOR(                DB11       ), K47XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K46XOR(                DB11       ), K54XOR(                DB12       ), K55XOR(                DB13       ), K18XOR(                DB14       ), K12XOR(                DB15       ), K34XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K08XOR((   1 & SALT) ? DB31 : DB15), K17XOR((   2 & SALT) ? DB00 : DB16), K21XOR((   4 & SALT) ? DB01 : DB17), K36XOR((   8 & SALT) ? DB02 : DB18), K23XOR((  16 & SALT) ? DB03 : DB19), K49XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K28XOR((  64 & SALT) ? DB03 : DB19), K15XOR(( 128 & SALT) ? DB04 : DB20), K24XOR(( 256 & SALT) ? DB05 : DB21), K37XOR(( 512 & SALT) ? DB06 : DB22), K07XOR((1024 & SALT) ? DB07 : DB23), K16XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K44XOR(                DB23       ), K09XOR(                DB24       ), K22XOR(                DB25       ), K42XOR(                DB26       ), K00XOR(                DB27       ), K10XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K30XOR(                DB27       ), K01XOR(                DB28       ), K02XOR(                DB29       ), K43XOR(                DB30       ), K35XOR(                DB31       ), K14XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(48);
		if (!threadIdx.y) {
		s1(K46XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K25XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K12XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K31XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K27XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K54XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K13XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K34XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K39XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K47XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K52XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K19XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K38XOR(db[39<<DB_SHIFT]), K11XOR(db[40<<DB_SHIFT]), K05XOR(db[41<<DB_SHIFT]), K06XOR(db[42<<DB_SHIFT]), K48XOR(db[43<<DB_SHIFT]), K33XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K32XOR(db[43<<DB_SHIFT]), K40XOR(db[44<<DB_SHIFT]), K41XOR(db[45<<DB_SHIFT]), K04XOR(db[46<<DB_SHIFT]), K53XOR(db[47<<DB_SHIFT]), K20XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K51XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K03XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K07XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K22XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K09XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K35XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K14XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K01XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K10XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K23XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K50XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K02XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K30XOR(db[55<<DB_SHIFT]), K24XOR(db[56<<DB_SHIFT]), K08XOR(db[57<<DB_SHIFT]), K28XOR(db[58<<DB_SHIFT]), K43XOR(db[59<<DB_SHIFT]), K49XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K16XOR(db[59<<DB_SHIFT]), K44XOR(db[60<<DB_SHIFT]), K17XOR(db[61<<DB_SHIFT]), K29XOR(db[62<<DB_SHIFT]), K21XOR(db[63<<DB_SHIFT]), K00XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(144);
		if (!threadIdx.y) {
		s1(K32XOR((   1 & SALT) ? DB15 : DB31), K11XOR((   2 & SALT) ? DB16 : DB00), K53XOR((   4 & SALT) ? DB17 : DB01), K48XOR((   8 & SALT) ? DB18 : DB02), K13XOR((  16 & SALT) ? DB19 : DB03), K40XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K54XOR((  64 & SALT) ? DB19 : DB03), K20XOR(( 128 & SALT) ? DB20 : DB04), K25XOR(( 256 & SALT) ? DB21 : DB05), K33XOR(( 512 & SALT) ? DB22 : DB06), K38XOR((1024 & SALT) ? DB23 : DB07), K05XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K55XOR(                DB07       ), K52XOR(                DB08       ), K46XOR(                DB09       ), K47XOR(                DB10       ), K34XOR(                DB11       ), K19XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K18XOR(                DB11       ), K26XOR(                DB12       ), K27XOR(                DB13       ), K45XOR(                DB14       ), K39XOR(                DB15       ), K06XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K37XOR((   1 & SALT) ? DB31 : DB15), K42XOR((   2 & SALT) ? DB00 : DB16), K50XOR((   4 & SALT) ? DB01 : DB17), K08XOR((   8 & SALT) ? DB02 : DB18), K24XOR((  16 & SALT) ? DB03 : DB19), K21XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K00XOR((  64 & SALT) ? DB03 : DB19), K44XOR(( 128 & SALT) ? DB04 : DB20), K49XOR(( 256 & SALT) ? DB05 : DB21), K09XOR(( 512 & SALT) ? DB06 : DB22), K36XOR((1024 & SALT) ? DB07 : DB23), K17XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K16XOR(                DB23       ), K10XOR(                DB24       ), K51XOR(                DB25       ), K14XOR(                DB26       ), K29XOR(                DB27       ), K35XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K02XOR(                DB27       ), K30XOR(                DB28       ), K03XOR(                DB29       ), K15XOR(                DB30       ), K07XOR(                DB31       ), K43XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(144);
		if (!threadIdx.y) {
		s1(K18XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K52XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K39XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K34XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K54XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K26XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K40XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K06XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K11XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K19XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K55XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K46XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K41XOR(db[39<<DB_SHIFT]), K38XOR(db[40<<DB_SHIFT]), K32XOR(db[41<<DB_SHIFT]), K33XOR(db[42<<DB_SHIFT]), K20XOR(db[43<<DB_SHIFT]), K05XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K04XOR(db[43<<DB_SHIFT]), K12XOR(db[44<<DB_SHIFT]), K13XOR(db[45<<DB_SHIFT]), K31XOR(db[46<<DB_SHIFT]), K25XOR(db[47<<DB_SHIFT]), K47XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K23XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K28XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K36XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K51XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K10XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K07XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K43XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K30XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K35XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K24XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K22XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K03XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K02XOR(db[55<<DB_SHIFT]), K49XOR(db[56<<DB_SHIFT]), K37XOR(db[57<<DB_SHIFT]), K00XOR(db[58<<DB_SHIFT]), K15XOR(db[59<<DB_SHIFT]), K21XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K17XOR(db[59<<DB_SHIFT]), K16XOR(db[60<<DB_SHIFT]), K42XOR(db[61<<DB_SHIFT]), K01XOR(db[62<<DB_SHIFT]), K50XOR(db[63<<DB_SHIFT]), K29XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(240);
		if (!threadIdx.y) {
		s1(K04XOR((   1 & SALT) ? DB15 : DB31), K38XOR((   2 & SALT) ? DB16 : DB00), K25XOR((   4 & SALT) ? DB17 : DB01), K20XOR((   8 & SALT) ? DB18 : DB02), K40XOR((  16 & SALT) ? DB19 : DB03), K12XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K26XOR((  64 & SALT) ? DB19 : DB03), K47XOR(( 128 & SALT) ? DB20 : DB04), K52XOR(( 256 & SALT) ? DB21 : DB05), K05XOR(( 512 & SALT) ? DB22 : DB06), K41XOR((1024 & SALT) ? DB23 : DB07), K32XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K27XOR(                DB07       ), K55XOR(                DB08       ), K18XOR(                DB09       ), K19XOR(                DB10       ), K06XOR(                DB11       ), K46XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K45XOR(                DB11       ), K53XOR(                DB12       ), K54XOR(                DB13       ), K48XOR(                DB14       ), K11XOR(                DB15       ), K33XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K09XOR((   1 & SALT) ? DB31 : DB15), K14XOR((   2 & SALT) ? DB00 : DB16), K22XOR((   4 & SALT) ? DB01 : DB17), K37XOR((   8 & SALT) ? DB02 : DB18), K49XOR((  16 & SALT) ? DB03 : DB19), K50XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K29XOR((  64 & SALT) ? DB03 : DB19), K16XOR(( 128 & SALT) ? DB04 : DB20), K21XOR(( 256 & SALT) ? DB05 : DB21), K10XOR(( 512 & SALT) ? DB06 : DB22), K08XOR((1024 & SALT) ? DB07 : DB23), K42XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K17XOR(                DB23       ), K35XOR(                DB24       ), K23XOR(                DB25       ), K43XOR(                DB26       ), K01XOR(                DB27       ), K07XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K03XOR(                DB27       ), K02XOR(                DB28       ), K28XOR(                DB29       ), K44XOR(                DB30       ), K36XOR(                DB31       ), K15XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(240);
		if (!threadIdx.y) {
		s1(K45XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K55XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K11XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K06XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K26XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K53XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K12XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K33XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K38XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K46XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K27XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K18XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K13XOR(db[39<<DB_SHIFT]), K41XOR(db[40<<DB_SHIFT]), K04XOR(db[41<<DB_SHIFT]), K05XOR(db[42<<DB_SHIFT]), K47XOR(db[43<<DB_SHIFT]), K32XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K31XOR(db[43<<DB_SHIFT]), K39XOR(db[44<<DB_SHIFT]), K40XOR(db[45<<DB_SHIFT]), K34XOR(db[46<<DB_SHIFT]), K52XOR(db[47<<DB_SHIFT]), K19XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K24XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K00XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K08XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K23XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K35XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K36XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K15XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K02XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K07XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K49XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K51XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K28XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K03XOR(db[55<<DB_SHIFT]), K21XOR(db[56<<DB_SHIFT]), K09XOR(db[57<<DB_SHIFT]), K29XOR(db[58<<DB_SHIFT]), K44XOR(db[59<<DB_SHIFT]), K50XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K42XOR(db[59<<DB_SHIFT]), K17XOR(db[60<<DB_SHIFT]), K14XOR(db[61<<DB_SHIFT]), K30XOR(db[62<<DB_SHIFT]), K22XOR(db[63<<DB_SHIFT]), K01XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(336);
		if (!threadIdx.y) {
		s1(K31XOR((   1 & SALT) ? DB15 : DB31), K41XOR((   2 & SALT) ? DB16 : DB00), K52XOR((   4 & SALT) ? DB17 : DB01), K47XOR((   8 & SALT) ? DB18 : DB02), K12XOR((  16 & SALT) ? DB19 : DB03), K39XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K53XOR((  64 & SALT) ? DB19 : DB03), K19XOR(( 128 & SALT) ? DB20 : DB04), K55XOR(( 256 & SALT) ? DB21 : DB05), K32XOR(( 512 & SALT) ? DB22 : DB06), K13XOR((1024 & SALT) ? DB23 : DB07), K04XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K54XOR(                DB07       ), K27XOR(                DB08       ), K45XOR(                DB09       ), K46XOR(                DB10       ), K33XOR(                DB11       ), K18XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K48XOR(                DB11       ), K25XOR(                DB12       ), K26XOR(                DB13       ), K20XOR(                DB14       ), K38XOR(                DB15       ), K05XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K10XOR((   1 & SALT) ? DB31 : DB15), K43XOR((   2 & SALT) ? DB00 : DB16), K51XOR((   4 & SALT) ? DB01 : DB17), K09XOR((   8 & SALT) ? DB02 : DB18), K21XOR((  16 & SALT) ? DB03 : DB19), K22XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K01XOR((  64 & SALT) ? DB03 : DB19), K17XOR(( 128 & SALT) ? DB04 : DB20), K50XOR(( 256 & SALT) ? DB05 : DB21), K35XOR(( 512 & SALT) ? DB06 : DB22), K37XOR((1024 & SALT) ? DB07 : DB23), K14XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K42XOR(                DB23       ), K07XOR(                DB24       ), K24XOR(                DB25       ), K15XOR(                DB26       ), K30XOR(                DB27       ), K36XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K28XOR(                DB27       ), K03XOR(                DB28       ), K00XOR(                DB29       ), K16XOR(                DB30       ), K08XOR(                DB31       ), K44XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(336);
		if (!threadIdx.y) {
		s1(K55XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K34XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K45XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K40XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K05XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K32XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K46XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K12XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K48XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K25XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K06XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K52XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K47XOR(db[39<<DB_SHIFT]), K20XOR(db[40<<DB_SHIFT]), K38XOR(db[41<<DB_SHIFT]), K39XOR(db[42<<DB_SHIFT]), K26XOR(db[43<<DB_SHIFT]), K11XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K41XOR(db[43<<DB_SHIFT]), K18XOR(db[44<<DB_SHIFT]), K19XOR(db[45<<DB_SHIFT]), K13XOR(db[46<<DB_SHIFT]), K31XOR(db[47<<DB_SHIFT]), K53XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K03XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K36XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K44XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K02XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K14XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K15XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K51XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K10XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K43XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K28XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K30XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K07XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K35XOR(db[55<<DB_SHIFT]), K00XOR(db[56<<DB_SHIFT]), K17XOR(db[57<<DB_SHIFT]), K08XOR(db[58<<DB_SHIFT]), K23XOR(db[59<<DB_SHIFT]), K29XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K21XOR(db[59<<DB_SHIFT]), K49XOR(db[60<<DB_SHIFT]), K50XOR(db[61<<DB_SHIFT]), K09XOR(db[62<<DB_SHIFT]), K01XOR(db[63<<DB_SHIFT]), K37XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(432);
		if (!threadIdx.y) {
		s1(K41XOR((   1 & SALT) ? DB15 : DB31), K20XOR((   2 & SALT) ? DB16 : DB00), K31XOR((   4 & SALT) ? DB17 : DB01), K26XOR((   8 & SALT) ? DB18 : DB02), K46XOR((  16 & SALT) ? DB19 : DB03), K18XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K32XOR((  64 & SALT) ? DB19 : DB03), K53XOR(( 128 & SALT) ? DB20 : DB04), K34XOR(( 256 & SALT) ? DB21 : DB05), K11XOR(( 512 & SALT) ? DB22 : DB06), K47XOR((1024 & SALT) ? DB23 : DB07), K38XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K33XOR(                DB07       ), K06XOR(                DB08       ), K55XOR(                DB09       ), K25XOR(                DB10       ), K12XOR(                DB11       ), K52XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K27XOR(                DB11       ), K04XOR(                DB12       ), K05XOR(                DB13       ), K54XOR(                DB14       ), K48XOR(                DB15       ), K39XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K42XOR((   1 & SALT) ? DB31 : DB15), K22XOR((   2 & SALT) ? DB00 : DB16), K30XOR((   4 & SALT) ? DB01 : DB17), K17XOR((   8 & SALT) ? DB02 : DB18), K00XOR((  16 & SALT) ? DB03 : DB19), K01XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K37XOR((  64 & SALT) ? DB03 : DB19), K49XOR(( 128 & SALT) ? DB04 : DB20), K29XOR(( 256 & SALT) ? DB05 : DB21), K14XOR(( 512 & SALT) ? DB06 : DB22), K16XOR((1024 & SALT) ? DB07 : DB23), K50XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K21XOR(                DB23       ), K43XOR(                DB24       ), K03XOR(                DB25       ), K51XOR(                DB26       ), K09XOR(                DB27       ), K15XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K07XOR(                DB27       ), K35XOR(                DB28       ), K36XOR(                DB29       ), K24XOR(                DB30       ), K44XOR(                DB31       ), K23XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(432);
		if (!threadIdx.y) {
		s1(K27XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K06XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K48XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K12XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K32XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K04XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K18XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K39XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K20XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K52XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K33XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K55XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K19XOR(db[39<<DB_SHIFT]), K47XOR(db[40<<DB_SHIFT]), K41XOR(db[41<<DB_SHIFT]), K11XOR(db[42<<DB_SHIFT]), K53XOR(db[43<<DB_SHIFT]), K38XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K13XOR(db[43<<DB_SHIFT]), K45XOR(db[44<<DB_SHIFT]), K46XOR(db[45<<DB_SHIFT]), K40XOR(db[46<<DB_SHIFT]), K34XOR(db[47<<DB_SHIFT]), K25XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K28XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K08XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K16XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K03XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K43XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K44XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K23XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K35XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K15XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K00XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K02XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K36XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K07XOR(db[55<<DB_SHIFT]), K29XOR(db[56<<DB_SHIFT]), K42XOR(db[57<<DB_SHIFT]), K37XOR(db[58<<DB_SHIFT]), K24XOR(db[59<<DB_SHIFT]), K01XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K50XOR(db[59<<DB_SHIFT]), K21XOR(db[60<<DB_SHIFT]), K22XOR(db[61<<DB_SHIFT]), K10XOR(db[62<<DB_SHIFT]), K30XOR(db[63<<DB_SHIFT]), K09XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(528);
		if (!threadIdx.y) {
		s1(K13XOR((   1 & SALT) ? DB15 : DB31), K47XOR((   2 & SALT) ? DB16 : DB00), K34XOR((   4 & SALT) ? DB17 : DB01), K53XOR((   8 & SALT) ? DB18 : DB02), K18XOR((  16 & SALT) ? DB19 : DB03), K45XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K04XOR((  64 & SALT) ? DB19 : DB03), K25XOR(( 128 & SALT) ? DB20 : DB04), K06XOR(( 256 & SALT) ? DB21 : DB05), K38XOR(( 512 & SALT) ? DB22 : DB06), K19XOR((1024 & SALT) ? DB23 : DB07), K41XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K05XOR(                DB07       ), K33XOR(                DB08       ), K27XOR(                DB09       ), K52XOR(                DB10       ), K39XOR(                DB11       ), K55XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K54XOR(                DB11       ), K31XOR(                DB12       ), K32XOR(                DB13       ), K26XOR(                DB14       ), K20XOR(                DB15       ), K11XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K14XOR((   1 & SALT) ? DB31 : DB15), K51XOR((   2 & SALT) ? DB00 : DB16), K02XOR((   4 & SALT) ? DB01 : DB17), K42XOR((   8 & SALT) ? DB02 : DB18), K29XOR((  16 & SALT) ? DB03 : DB19), K30XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K09XOR((  64 & SALT) ? DB03 : DB19), K21XOR(( 128 & SALT) ? DB04 : DB20), K01XOR(( 256 & SALT) ? DB05 : DB21), K43XOR(( 512 & SALT) ? DB06 : DB22), K17XOR((1024 & SALT) ? DB07 : DB23), K22XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K50XOR(                DB23       ), K15XOR(                DB24       ), K28XOR(                DB25       ), K23XOR(                DB26       ), K10XOR(                DB27       ), K44XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K36XOR(                DB27       ), K07XOR(                DB28       ), K08XOR(                DB29       ), K49XOR(                DB30       ), K16XOR(                DB31       ), K24XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(528);
		if (!threadIdx.y) {
		s1(K54XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K33XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K20XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K39XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K04XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K31XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K45XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K11XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K47XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K55XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K05XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K27XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K46XOR(db[39<<DB_SHIFT]), K19XOR(db[40<<DB_SHIFT]), K13XOR(db[41<<DB_SHIFT]), K38XOR(db[42<<DB_SHIFT]), K25XOR(db[43<<DB_SHIFT]), K41XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K40XOR(db[43<<DB_SHIFT]), K48XOR(db[44<<DB_SHIFT]), K18XOR(db[45<<DB_SHIFT]), K12XOR(db[46<<DB_SHIFT]), K06XOR(db[47<<DB_SHIFT]), K52XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K00XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K37XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K17XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K28XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K15XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K16XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K24XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K07XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K44XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K29XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K03XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K08XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K36XOR(db[55<<DB_SHIFT]), K01XOR(db[56<<DB_SHIFT]), K14XOR(db[57<<DB_SHIFT]), K09XOR(db[58<<DB_SHIFT]), K49XOR(db[59<<DB_SHIFT]), K30XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K22XOR(db[59<<DB_SHIFT]), K50XOR(db[60<<DB_SHIFT]), K51XOR(db[61<<DB_SHIFT]), K35XOR(db[62<<DB_SHIFT]), K02XOR(db[63<<DB_SHIFT]), K10XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(624);
		if (!threadIdx.y) {
		s1(K40XOR((   1 & SALT) ? DB15 : DB31), K19XOR((   2 & SALT) ? DB16 : DB00), K06XOR((   4 & SALT) ? DB17 : DB01), K25XOR((   8 & SALT) ? DB18 : DB02), K45XOR((  16 & SALT) ? DB19 : DB03), K48XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K31XOR((  64 & SALT) ? DB19 : DB03), K52XOR(( 128 & SALT) ? DB20 : DB04), K33XOR(( 256 & SALT) ? DB21 : DB05), K41XOR(( 512 & SALT) ? DB22 : DB06), K46XOR((1024 & SALT) ? DB23 : DB07), K13XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K32XOR(                DB07       ), K05XOR(                DB08       ), K54XOR(                DB09       ), K55XOR(                DB10       ), K11XOR(                DB11       ), K27XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K26XOR(                DB11       ), K34XOR(                DB12       ), K04XOR(                DB13       ), K53XOR(                DB14       ), K47XOR(                DB15       ), K38XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K43XOR((   1 & SALT) ? DB31 : DB15), K23XOR((   2 & SALT) ? DB00 : DB16), K03XOR((   4 & SALT) ? DB01 : DB17), K14XOR((   8 & SALT) ? DB02 : DB18), K01XOR((  16 & SALT) ? DB03 : DB19), K02XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K10XOR((  64 & SALT) ? DB03 : DB19), K50XOR(( 128 & SALT) ? DB04 : DB20), K30XOR(( 256 & SALT) ? DB05 : DB21), K15XOR(( 512 & SALT) ? DB06 : DB22), K42XOR((1024 & SALT) ? DB07 : DB23), K51XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K22XOR(                DB23       ), K44XOR(                DB24       ), K00XOR(                DB25       ), K24XOR(                DB26       ), K35XOR(                DB27       ), K16XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K08XOR(                DB27       ), K36XOR(                DB28       ), K37XOR(                DB29       ), K21XOR(                DB30       ), K17XOR(                DB31       ), K49XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		

		// ROUND_B(624);
		if (!threadIdx.y) {
		s1(K26XOR((   1 & SALT) ? db[47<<DB_SHIFT] : db[63<<DB_SHIFT]), K05XOR((   2 & SALT) ? db[48<<DB_SHIFT] : db[32<<DB_SHIFT]), K47XOR((   4 & SALT) ? db[49<<DB_SHIFT] : db[33<<DB_SHIFT]), K11XOR((   8 & SALT) ? db[50<<DB_SHIFT] : db[34<<DB_SHIFT]), K31XOR((  16 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K34XOR((  32 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), DB08, DB16, DB22, DB30);
		s2(K48XOR((  64 & SALT) ? db[51<<DB_SHIFT] : db[35<<DB_SHIFT]), K38XOR(( 128 & SALT) ? db[52<<DB_SHIFT] : db[36<<DB_SHIFT]), K19XOR(( 256 & SALT) ? db[53<<DB_SHIFT] : db[37<<DB_SHIFT]), K27XOR(( 512 & SALT) ? db[54<<DB_SHIFT] : db[38<<DB_SHIFT]), K32XOR((1024 & SALT) ? db[55<<DB_SHIFT] : db[39<<DB_SHIFT]), K54XOR((2048 & SALT) ? db[56<<DB_SHIFT] : db[40<<DB_SHIFT]), DB12, DB27, DB01, DB17);
		s3(K18XOR(db[39<<DB_SHIFT]), K46XOR(db[40<<DB_SHIFT]), K40XOR(db[41<<DB_SHIFT]), K41XOR(db[42<<DB_SHIFT]), K52XOR(db[43<<DB_SHIFT]), K13XOR(db[44<<DB_SHIFT]), DB23, DB15, DB29, DB05);
		s4(K12XOR(db[43<<DB_SHIFT]), K20XOR(db[44<<DB_SHIFT]), K45XOR(db[45<<DB_SHIFT]), K39XOR(db[46<<DB_SHIFT]), K33XOR(db[47<<DB_SHIFT]), K55XOR(db[48<<DB_SHIFT]), DB25, DB19, DB09, DB00);
		s5(K29XOR((   1 & SALT) ? db[63<<DB_SHIFT] : db[47<<DB_SHIFT]), K09XOR((   2 & SALT) ? db[32<<DB_SHIFT] : db[48<<DB_SHIFT]), K42XOR((   4 & SALT) ? db[33<<DB_SHIFT] : db[49<<DB_SHIFT]), K00XOR((   8 & SALT) ? db[34<<DB_SHIFT] : db[50<<DB_SHIFT]), K44XOR((  16 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K17XOR((  32 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), DB07, DB13, DB24, DB02);
		s6(K49XOR((  64 & SALT) ? db[35<<DB_SHIFT] : db[51<<DB_SHIFT]), K36XOR(( 128 & SALT) ? db[36<<DB_SHIFT] : db[52<<DB_SHIFT]), K16XOR(( 256 & SALT) ? db[37<<DB_SHIFT] : db[53<<DB_SHIFT]), K01XOR(( 512 & SALT) ? db[38<<DB_SHIFT] : db[54<<DB_SHIFT]), K28XOR((1024 & SALT) ? db[39<<DB_SHIFT] : db[55<<DB_SHIFT]), K37XOR((2048 & SALT) ? db[40<<DB_SHIFT] : db[56<<DB_SHIFT]), DB03, DB28, DB10, DB18);
		s7(K08XOR(db[55<<DB_SHIFT]), K30XOR(db[56<<DB_SHIFT]), K43XOR(db[57<<DB_SHIFT]), K10XOR(db[58<<DB_SHIFT]), K21XOR(db[59<<DB_SHIFT]), K02XOR(db[60<<DB_SHIFT]), DB31, DB11, DB21, DB06);
		s8(K51XOR(db[59<<DB_SHIFT]), K22XOR(db[60<<DB_SHIFT]), K23XOR(db[61<<DB_SHIFT]), K07XOR(db[62<<DB_SHIFT]), K03XOR(db[63<<DB_SHIFT]), K35XOR(db[32<<DB_SHIFT]), DB04, DB26, DB14, DB20);
		}
		

		// ROUND_A(720);
		if (!threadIdx.y) {
		s1(K19XOR((   1 & SALT) ? DB15 : DB31), K53XOR((   2 & SALT) ? DB16 : DB00), K40XOR((   4 & SALT) ? DB17 : DB01), K04XOR((   8 & SALT) ? DB18 : DB02), K55XOR((  16 & SALT) ? DB19 : DB03), K27XOR((  32 & SALT) ? DB20 : DB04), db[40<<DB_SHIFT], db[48<<DB_SHIFT], db[54<<DB_SHIFT], db[62<<DB_SHIFT]);
		s2(K41XOR((  64 & SALT) ? DB19 : DB03), K31XOR(( 128 & SALT) ? DB20 : DB04), K12XOR(( 256 & SALT) ? DB21 : DB05), K20XOR(( 512 & SALT) ? DB22 : DB06), K25XOR((1024 & SALT) ? DB23 : DB07), K47XOR((2048 & SALT) ? DB24 : DB08), db[44<<DB_SHIFT], db[59<<DB_SHIFT], db[33<<DB_SHIFT], db[49<<DB_SHIFT]);
		s3(K11XOR(                DB07       ), K39XOR(                DB08       ), K33XOR(                DB09       ), K34XOR(                DB10       ), K45XOR(                DB11       ), K06XOR(                DB12       ), db[55<<DB_SHIFT], db[47<<DB_SHIFT], db[61<<DB_SHIFT], db[37<<DB_SHIFT]);
		s4(K05XOR(                DB11       ), K13XOR(                DB12       ), K38XOR(                DB13       ), K32XOR(                DB14       ), K26XOR(                DB15       ), K48XOR(                DB16       ), db[57<<DB_SHIFT], db[51<<DB_SHIFT], db[41<<DB_SHIFT], db[32<<DB_SHIFT]);
		s5(K22XOR((   1 & SALT) ? DB31 : DB15), K02XOR((   2 & SALT) ? DB00 : DB16), K35XOR((   4 & SALT) ? DB01 : DB17), K50XOR((   8 & SALT) ? DB02 : DB18), K37XOR((  16 & SALT) ? DB03 : DB19), K10XOR((  32 & SALT) ? DB04 : DB20), db[39<<DB_SHIFT], db[45<<DB_SHIFT], db[56<<DB_SHIFT], db[34<<DB_SHIFT]);
		s6(K42XOR((  64 & SALT) ? DB03 : DB19), K29XOR(( 128 & SALT) ? DB04 : DB20), K09XOR(( 256 & SALT) ? DB05 : DB21), K51XOR(( 512 & SALT) ? DB06 : DB22), K21XOR((1024 & SALT) ? DB07 : DB23), K30XOR((2048 & SALT) ? DB08 : DB24), db[35<<DB_SHIFT], db[60<<DB_SHIFT], db[42<<DB_SHIFT], db[50<<DB_SHIFT]);
		s7(K01XOR(                DB23       ), K23XOR(                DB24       ), K36XOR(                DB25       ), K03XOR(                DB26       ), K14XOR(                DB27       ), K24XOR(                DB28       ), db[63<<DB_SHIFT], db[43<<DB_SHIFT], db[53<<DB_SHIFT], db[38<<DB_SHIFT]);
		s8(K44XOR(                DB27       ), K15XOR(                DB28       ), K16XOR(                DB29       ), K00XOR(                DB30       ), K49XOR(                DB31       ), K28XOR(                DB00       ), db[36<<DB_SHIFT], db[58<<DB_SHIFT], db[46<<DB_SHIFT], db[52<<DB_SHIFT]);
		}
		
	}
	db[ 0<<DB_SHIFT] = DB00;
	db[ 1<<DB_SHIFT] = DB01;
	db[ 2<<DB_SHIFT] = DB02;
	db[ 3<<DB_SHIFT] = DB03;
	db[ 4<<DB_SHIFT] = DB04;
	db[ 5<<DB_SHIFT] = DB05;
	db[ 6<<DB_SHIFT] = DB06;
	db[ 7<<DB_SHIFT] = DB07;
	db[ 8<<DB_SHIFT] = DB08;
	db[ 9<<DB_SHIFT] = DB09;
	db[10<<DB_SHIFT] = DB10;
	db[11<<DB_SHIFT] = DB11;
	db[12<<DB_SHIFT] = DB12;
	db[13<<DB_SHIFT] = DB13;
	db[14<<DB_SHIFT] = DB14;
	db[15<<DB_SHIFT] = DB15;
	db[16<<DB_SHIFT] = DB16;
	db[17<<DB_SHIFT] = DB17;
	db[18<<DB_SHIFT] = DB18;
	db[19<<DB_SHIFT] = DB19;
	db[20<<DB_SHIFT] = DB20;
	db[21<<DB_SHIFT] = DB21;
	db[22<<DB_SHIFT] = DB22;
	db[23<<DB_SHIFT] = DB23;
	db[24<<DB_SHIFT] = DB24;
	db[25<<DB_SHIFT] = DB25;
	db[26<<DB_SHIFT] = DB26;
	db[27<<DB_SHIFT] = DB27;
	db[28<<DB_SHIFT] = DB28;
	db[29<<DB_SHIFT] = DB29;
	db[30<<DB_SHIFT] = DB30;
	db[31<<DB_SHIFT] = DB31;
}

#define GET_TRIPCODE_CHAR_INDEX(r, t, i0, i1, i2, i3, i4, i5, pos)  \
		(  ((((r)[threadIdx.x + (i0<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (5 + ((pos) * 6)))  \
	 	 | ((((r)[threadIdx.x + (i1<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (4 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i2<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (3 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i3<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (2 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i4<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (1 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i5<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (0 + ((pos) * 6)))) \

#define GET_TRIPCODE_CHAR_INDEX_LAST(r, t, i0, i1, i2, i3)     \
		(  ((((r)[threadIdx.x + (i0<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 5)  \
	 	 | ((((r)[threadIdx.x + (i1<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 4)  \
		 | ((((r)[threadIdx.x + (i2<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 3)  \
		 | ((((r)[threadIdx.x + (i3<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 2)) \

DES_FUNCTION_QUALIFIERS void
DES_GetTripcodeChunks(int tripcodeIndex, unsigned int *tripcodeChunkArray, int searchMode)
{
	// Perform the final permutation here.
	if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
	} else if (searchMode == SEARCH_MODE_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
		                        | GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
								| GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] = ((tripcodeChunkArray[0] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0);
		tripcodeChunkArray[2] = ((tripcodeChunkArray[1] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0);
		tripcodeChunkArray[3] = ((tripcodeChunkArray[2] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0);
		tripcodeChunkArray[4] = ((tripcodeChunkArray[3] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0);
		tripcodeChunkArray[5] = ((tripcodeChunkArray[4] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	}
}

DES_FUNCTION_QUALIFIERS
unsigned char *DES_GetTripcode(int tripcodeIndex, unsigned char *tripcode)
{
	// Perform the final permutation as necessary.
  	tripcode[0] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0)];
  	tripcode[1] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0)];
  	tripcode[2] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0)];
  	tripcode[3] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0)];
  	tripcode[4] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0)];
  	tripcode[5] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0)];
  	tripcode[6] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0)];
  	tripcode[7] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0)];
  	tripcode[8] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0)];
	tripcode[9] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24)];
 	tripcode[10] = '\0';

	return tripcode;
}

#define SET_KEY_CHAR(var, flag, table, value)             \
	if (!(flag)) {                                        \
		var = (table)[(value)];                           \
		isSecondByte = IS_FIRST_BYTE_SJIS(var);           \
	} else {                                              \
		var = CUDA_keyCharTable_SecondByte[(value)];          \
		isSecondByte = FALSE;                             \
	}

#define CUDA_DES_DEFINE_SEARCH_FUNCTION(functionName) \
__global__ void functionName(\
	GPUOutput *outputArray,\
	unsigned char      *keyBitmap,\
	unsigned int     *tripcodeChunkArray,\
	unsigned int      numTripcodeChunk,\
	unsigned int keyFrom00To27,\
	int          intSalt,\
	int         searchMode) {

#define CUDA_DES_BEFORE_SEARCHING \
	GPUOutput  *output = &outputArray[blockIdx.x * CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK + threadIdx.x];\
	unsigned char        key[8];\
	BOOL         isSecondByte;\
	unsigned char        tripcodeIndex;\
	unsigned char        passCount;\
	\
	if (threadIdx.y == 0) {\
		output->numMatchingTripcodes = 0;\
	}\
	key[0] = CUDA_key[0];\
	key[1] = CUDA_key[1];\
	key[2] = CUDA_key[2];\
	key[3] = CUDA_key[3];\
	isSecondByte = IS_FIRST_BYTE_SJIS(CUDA_key[3]);\
	SET_KEY_CHAR(key[4], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[4] + ((blockIdx.x  >> 6) & 63));\
	SET_KEY_CHAR(key[5], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[5] + ( blockIdx.x        & 63));\
	SET_KEY_CHAR(key[6], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[6] + ( threadIdx.x       & 63));\
	unsigned int keyFrom28To48 = (((unsigned int)key[6] & 0x7f) << 14) | (((unsigned int)key[5] & 0x7f) <<  7) | (((unsigned int)key[4] & 0x7f) << 0); \
	\
	for (passCount = 0; passCount < 1; ++passCount) {\
	__syncthreads();\
	DES_Crypt(keyFrom00To27, keyFrom28To48, intSalt);\
	\
	__syncthreads();\
	if (threadIdx.y == 0) {\
		BOOL found = FALSE;\
		for (tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {

#define CUDA_DES_END_OF_SEAERCH_FUNCTION \
		}\
quit_loops:\
		if (found == TRUE) {\
			output->numMatchingTripcodes  = 1;\
			output->pair.key.c[0] = key[0];\
			output->pair.key.c[1] = key[1];\
			output->pair.key.c[2] = key[2];\
			output->pair.key.c[3] = key[3];\
			output->pair.key.c[4] = key[4];\
			output->pair.key.c[5] = key[5];\
			output->pair.key.c[6] = key[6];\
			output->pair.key.c[7] = CUDA_key7Array[tripcodeIndex];\
		}\
	}\
	}\
	if (threadIdx.y == 0)\
		output->numGeneratedTripcodes = CUDA_DES_BS_DEPTH;\
}

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == tripcodeChunk) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	if (keyBitmap[tripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)])
		continue;
	for (int j = 0; j < numTripcodeChunk; ++j){
		int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
		while (tripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
			middle = (lower + upper) >> 1;
			if (tripcodeChunk > tripcodeChunkArray[middle]) {
				lower = middle + 1;
			} else {
				upper = middle - 1;
			}
		}
		if (tripcodeChunk == tripcodeChunkArray[middle]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_BackwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[pos]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		unsigned int generatedTripcodeChunk = generatedTripcodeChunkArray[pos];
		if (keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)])
			continue;
		for (int j = 0; j < numTripcodeChunk; ++j){
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[0]) {
			found = TRUE;
			goto quit_loops;
		}
	}
	//
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[1]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching)
	unsigned int generatedTripcodeChunkArray[6];
	unsigned int generatedTripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[0];
	if (!keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[1];
	if (!keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j) {
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION



///////////////////////////////////////////////////////////////////////////////
// CUDA SEARCH THREAD FOR 10 CHARACTER TRIPCODES                             //
///////////////////////////////////////////////////////////////////////////////

#define SET_BIT_FOR_KEY7(var, k) if (key7 & (0x1 << (k))) (var) |= 0x1 << tripcodeIndex

unsigned WINAPI Thread_SearchForDESTripcodesOnCUDADevice(LPVOID info)
{
	hipError_t     hipError_t;
	hipDeviceProp_t  CUDADeviceProperties;
	unsigned int    numBlocksPerSM;
	unsigned int    numBlocksPerGrid;
	GPUOutput      *outputArray = NULL;
	GPUOutput      *CUDA_outputArray = NULL;
	unsigned int   *CUDA_tripcodeChunkArray = NULL;
	unsigned char  *CUDA_keyBitmap = NULL;
	unsigned int    sizeOutputArray;
	unsigned char   key[MAX_LEN_TRIPCODE + 1];
	unsigned char   salt[3];
	unsigned char   expansionFunction[96];
	char            status[LEN_LINE_BUFFER_FOR_SCREEN] = "";
	int             optimizationPhase    = CUDA_OPTIMIZATION_PHASE_NUM_BLOCKS;
	int             optimizationSubphase = 0;
	double          timeElapsedInOptimizationSubphase = 0;
	static int      numBlocksTableForOptimization[] = {8, 16, 32, 48, 64, 96, 128, 160, 192, 224, 256, 0};
	double          numGeneratedTripcodes = 0;
	double          speedInPreviousSubphase = 0;
	double          speedInCurrentSubphase = 0;
	DWORD           startingTime;
	DWORD           endingTime;
	double          deltaTime;

	key[lenTripcode] = '\0';
	salt[2]          = '\0';
	
	CUDA_ERROR(hipSetDevice(((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	CUDA_ERROR(hipGetDeviceProperties(&CUDADeviceProperties, ((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	if (CUDADeviceProperties.computeMode == hipComputeModeProhibited) {
		sprintf(status, "[disabled]");
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), FALSE, status);
		return 0;
	}

	if (options.CUDANumBlocksPerSM == CUDA_NUM_BLOCKS_PER_SM_NIL) {
		numBlocksPerSM = numBlocksTableForOptimization[optimizationSubphase];
	} else {
		numBlocksPerSM = options.CUDANumBlocksPerSM;
	}
	numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
	sizeOutputArray = CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK * numBlocksPerGrid;
	outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(outputArray == NULL, ERROR_NO_MEMORY, "Not enough memory.");
	hipError_t = hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);
	hipError_t = hipMalloc((void **)&CUDA_keyBitmap, KEY_BITMAP_SIZE);
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);
	hipError_t = hipMalloc((void **)&CUDA_tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk); 
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);

	CUDA_ERROR(hipMemcpy(CUDA_tripcodeChunkArray, tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpy(CUDA_keyBitmap, keyBitmap, KEY_BITMAP_SIZE, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	
	startingTime = timeGetTime();

	while (!GetTerminationState()) {
		// Choose the first 4 characters of the key.
		SetCharactersInTripcodeKey(key, 4);
		
		// Make sure that the first 3 bytes consist of valid Shift-JIS characters.
		for (int i = 4; i < lenTripcode; ++i)
			key[i] = 'A';
		if (!IsValidKey(key))
			continue;
		unsigned int keyFrom00To27 = (((unsigned int)key[3] & 0x7f) << 21) | (((unsigned int)key[2] & 0x7f) << 14) | (((unsigned int)key[1] & 0x7f) << 7) | (((unsigned int)key[0] & 0x7f) << 0); \
		
		// Generate random bytes for the key to ensure the randomness of them.
		unsigned char randomByteForKey6 = RandomByte();
		for (int i = 4; i < lenTripcode; ++i)
			key[i] = RandomByte();
		unsigned char key7Array[CUDA_DES_BS_DEPTH];
		DES_Vector  keyFrom49To55Array[7] = {0, 0, 0, 0, 0, 0, 0};
		for (int tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {
			unsigned char key7 = key7Array[tripcodeIndex] = keyCharTable_SecondByteAndOneByte[key[7] + tripcodeIndex];
			SET_BIT_FOR_KEY7(keyFrom49To55Array[0], 0);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[1], 1);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[2], 2);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[3], 3);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[4], 4);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[5], 5);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[6], 6);
		}

		// Create an expansion function based on the salt.
		salt[0] = CONVERT_CHAR_FOR_SALT(key[1]);
		salt[1] = CONVERT_CHAR_FOR_SALT(key[2]);
		int intSalt = charToIndexTableForDES[CONVERT_CHAR_FOR_SALT(key[1])] | (charToIndexTableForDES[CONVERT_CHAR_FOR_SALT(key[2])] << 6);
		DES_CreateExpansionFunction((char *)salt, expansionFunction);

		// Call an appropriate CUDA kernel.
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcode));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(expansionFunction)));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(key7Array)));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(keyFrom49To55Array)));
		dim3 dimBlock(CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK, CUDA_DES_NUM_THREADS_FOR_BITSLICE);
		dim3 dimGrid(numBlocksPerGrid);
		if (searchMode == SEARCH_MODE_FLEXIBLE) {
			/*if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_Flexible_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			} else {*/
				CUDA_PerformSearching_DES_Flexible<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			//}
		} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
			/*if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			} else {*/
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			//}
		} else {
			/*
			if (numTripcodeChunk == 1) {
				if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
					CUDA_PerformSearching_DES_ForwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
						CUDA_outputArray,
						CUDA_keyBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						keyFrom00To27,
						intSalt,
						searchMode);
				} else {
					CUDA_PerformSearching_DES_BackwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
						CUDA_outputArray,
						CUDA_keyBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						keyFrom00To27,
						intSalt,
						searchMode);
				}
			} else if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			} else {*/
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			//}
		}
		CUDA_ERROR(hipGetLastError());
		// CUDA_ERROR(hipDeviceSynchronize()); // Check errors at kernel launch.

		// Process the output array.
		CUDA_ERROR(hipMemcpy(outputArray, CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray, hipMemcpyDeviceToHost));
		// We can save registers this way. Not particularly safe, though.
		for (unsigned int indexOutput = 0; indexOutput < sizeOutputArray; indexOutput++){
			GPUOutput *output = &outputArray[indexOutput];
			if (output->numMatchingTripcodes > 0)
				GenerateDESTripcode(output->pair.tripcode.c, output->pair.key.c);
		}
		numGeneratedTripcodes += ProcessGPUOutput(key, outputArray, sizeOutputArray, FALSE);
		
		// Optimization
		endingTime = timeGetTime();
		deltaTime = (endingTime >= startingTime)
						? ((double)endingTime - (double)startingTime                     ) * 0.001
						: ((double)endingTime - (double)startingTime + (double)0xffffffff) * 0.001;
		while (GetPauseState() && !GetTerminationState())
			Sleep(PAUSE_INTERVAL);
		startingTime = timeGetTime();
		timeElapsedInOptimizationSubphase += deltaTime;
		speedInCurrentSubphase = numGeneratedTripcodes / timeElapsedInOptimizationSubphase;
		//
		if (optimizationPhase == CUDA_OPTIMIZATION_PHASE_NUM_BLOCKS) {
			if (options.CUDANumBlocksPerSM != CUDA_NUM_BLOCKS_PER_SM_NIL) {
				optimizationPhase     = CUDA_OPTIMIZATION_PHASE_COMPLETED;
				optimizationSubphase  = 0;
				numGeneratedTripcodes = 0;
				timeElapsedInOptimizationSubphase = 0;
			} else if (timeElapsedInOptimizationSubphase >= CUDA_OPTIMIZATION_SUBPHASE_DURATION) {
				if (   optimizationSubphase > 0
				    && (   speedInPreviousSubphase > speedInCurrentSubphase
					    || fabs(speedInPreviousSubphase - speedInCurrentSubphase) / speedInPreviousSubphase < CUDA_OPTIMIZATION_THRESHOLD)) {
					numBlocksPerSM = numBlocksTableForOptimization[(speedInPreviousSubphase > speedInCurrentSubphase) ? (optimizationSubphase - 1) : (optimizationSubphase)];
					optimizationPhase = CUDA_OPTIMIZATION_PHASE_COMPLETED;
					optimizationSubphase = 0;
					numGeneratedTripcodes = 0;
				} else if (numBlocksTableForOptimization[optimizationSubphase + 1] > 0) {
					numBlocksPerSM = numBlocksTableForOptimization[++optimizationSubphase];
					timeElapsedInOptimizationSubphase = 0;
					numGeneratedTripcodes = 0;
					speedInPreviousSubphase = speedInCurrentSubphase;
				} else {
					optimizationPhase = CUDA_OPTIMIZATION_PHASE_COMPLETED;
					optimizationSubphase = 0;
					numGeneratedTripcodes = 0;
				}
				timeElapsedInOptimizationSubphase = 0;
				numGeneratedTripcodes = 0;
				numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
				sizeOutputArray = CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK * numBlocksPerGrid;
				free(outputArray);
				outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
				ERROR0(outputArray == NULL, ERROR_NO_MEMORY, "Not enough memory.");
				CUDA_ERROR(hipFree(CUDA_outputArray));
				hipError_t = hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray);
				ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
				CUDA_ERROR(hipError_t);
			}
		}
		//
		sprintf(status,
			    "%s%.1lfM TPS, %d blocks/SM",
				(optimizationPhase != CUDA_OPTIMIZATION_PHASE_COMPLETED) ? "[optimizing...] " : "",
				speedInCurrentSubphase / 1000000,
				numBlocksPerSM);
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), (optimizationPhase != CUDA_OPTIMIZATION_PHASE_COMPLETED), status);
	}

	RELEASE_AND_SET_TO_NULL(CUDA_outputArray,        hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_tripcodeChunkArray, hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_keyBitmap,          hipFree);
	RELEASE_AND_SET_TO_NULL(outputArray,             free);
}
