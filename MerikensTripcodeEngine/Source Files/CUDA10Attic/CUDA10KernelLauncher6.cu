// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 3072
#include "CUDA10Kernel.h"
#define SALT 3073
#include "CUDA10Kernel.h"
#define SALT 3074
#include "CUDA10Kernel.h"
#define SALT 3075
#include "CUDA10Kernel.h"
#define SALT 3076
#include "CUDA10Kernel.h"
#define SALT 3077
#include "CUDA10Kernel.h"
#define SALT 3078
#include "CUDA10Kernel.h"
#define SALT 3079
#include "CUDA10Kernel.h"
#define SALT 3080
#include "CUDA10Kernel.h"
#define SALT 3081
#include "CUDA10Kernel.h"
#define SALT 3082
#include "CUDA10Kernel.h"
#define SALT 3083
#include "CUDA10Kernel.h"
#define SALT 3084
#include "CUDA10Kernel.h"
#define SALT 3085
#include "CUDA10Kernel.h"
#define SALT 3086
#include "CUDA10Kernel.h"
#define SALT 3087
#include "CUDA10Kernel.h"
#define SALT 3088
#include "CUDA10Kernel.h"
#define SALT 3089
#include "CUDA10Kernel.h"
#define SALT 3090
#include "CUDA10Kernel.h"
#define SALT 3091
#include "CUDA10Kernel.h"
#define SALT 3092
#include "CUDA10Kernel.h"
#define SALT 3093
#include "CUDA10Kernel.h"
#define SALT 3094
#include "CUDA10Kernel.h"
#define SALT 3095
#include "CUDA10Kernel.h"
#define SALT 3096
#include "CUDA10Kernel.h"
#define SALT 3097
#include "CUDA10Kernel.h"
#define SALT 3098
#include "CUDA10Kernel.h"
#define SALT 3099
#include "CUDA10Kernel.h"
#define SALT 3100
#include "CUDA10Kernel.h"
#define SALT 3101
#include "CUDA10Kernel.h"
#define SALT 3102
#include "CUDA10Kernel.h"
#define SALT 3103
#include "CUDA10Kernel.h"
#define SALT 3104
#include "CUDA10Kernel.h"
#define SALT 3105
#include "CUDA10Kernel.h"
#define SALT 3106
#include "CUDA10Kernel.h"
#define SALT 3107
#include "CUDA10Kernel.h"
#define SALT 3108
#include "CUDA10Kernel.h"
#define SALT 3109
#include "CUDA10Kernel.h"
#define SALT 3110
#include "CUDA10Kernel.h"
#define SALT 3111
#include "CUDA10Kernel.h"
#define SALT 3112
#include "CUDA10Kernel.h"
#define SALT 3113
#include "CUDA10Kernel.h"
#define SALT 3114
#include "CUDA10Kernel.h"
#define SALT 3115
#include "CUDA10Kernel.h"
#define SALT 3116
#include "CUDA10Kernel.h"
#define SALT 3117
#include "CUDA10Kernel.h"
#define SALT 3118
#include "CUDA10Kernel.h"
#define SALT 3119
#include "CUDA10Kernel.h"
#define SALT 3120
#include "CUDA10Kernel.h"
#define SALT 3121
#include "CUDA10Kernel.h"
#define SALT 3122
#include "CUDA10Kernel.h"
#define SALT 3123
#include "CUDA10Kernel.h"
#define SALT 3124
#include "CUDA10Kernel.h"
#define SALT 3125
#include "CUDA10Kernel.h"
#define SALT 3126
#include "CUDA10Kernel.h"
#define SALT 3127
#include "CUDA10Kernel.h"
#define SALT 3128
#include "CUDA10Kernel.h"
#define SALT 3129
#include "CUDA10Kernel.h"
#define SALT 3130
#include "CUDA10Kernel.h"
#define SALT 3131
#include "CUDA10Kernel.h"
#define SALT 3132
#include "CUDA10Kernel.h"
#define SALT 3133
#include "CUDA10Kernel.h"
#define SALT 3134
#include "CUDA10Kernel.h"
#define SALT 3135
#include "CUDA10Kernel.h"
#define SALT 3136
#include "CUDA10Kernel.h"
#define SALT 3137
#include "CUDA10Kernel.h"
#define SALT 3138
#include "CUDA10Kernel.h"
#define SALT 3139
#include "CUDA10Kernel.h"
#define SALT 3140
#include "CUDA10Kernel.h"
#define SALT 3141
#include "CUDA10Kernel.h"
#define SALT 3142
#include "CUDA10Kernel.h"
#define SALT 3143
#include "CUDA10Kernel.h"
#define SALT 3144
#include "CUDA10Kernel.h"
#define SALT 3145
#include "CUDA10Kernel.h"
#define SALT 3146
#include "CUDA10Kernel.h"
#define SALT 3147
#include "CUDA10Kernel.h"
#define SALT 3148
#include "CUDA10Kernel.h"
#define SALT 3149
#include "CUDA10Kernel.h"
#define SALT 3150
#include "CUDA10Kernel.h"
#define SALT 3151
#include "CUDA10Kernel.h"
#define SALT 3152
#include "CUDA10Kernel.h"
#define SALT 3153
#include "CUDA10Kernel.h"
#define SALT 3154
#include "CUDA10Kernel.h"
#define SALT 3155
#include "CUDA10Kernel.h"
#define SALT 3156
#include "CUDA10Kernel.h"
#define SALT 3157
#include "CUDA10Kernel.h"
#define SALT 3158
#include "CUDA10Kernel.h"
#define SALT 3159
#include "CUDA10Kernel.h"
#define SALT 3160
#include "CUDA10Kernel.h"
#define SALT 3161
#include "CUDA10Kernel.h"
#define SALT 3162
#include "CUDA10Kernel.h"
#define SALT 3163
#include "CUDA10Kernel.h"
#define SALT 3164
#include "CUDA10Kernel.h"
#define SALT 3165
#include "CUDA10Kernel.h"
#define SALT 3166
#include "CUDA10Kernel.h"
#define SALT 3167
#include "CUDA10Kernel.h"
#define SALT 3168
#include "CUDA10Kernel.h"
#define SALT 3169
#include "CUDA10Kernel.h"
#define SALT 3170
#include "CUDA10Kernel.h"
#define SALT 3171
#include "CUDA10Kernel.h"
#define SALT 3172
#include "CUDA10Kernel.h"
#define SALT 3173
#include "CUDA10Kernel.h"
#define SALT 3174
#include "CUDA10Kernel.h"
#define SALT 3175
#include "CUDA10Kernel.h"
#define SALT 3176
#include "CUDA10Kernel.h"
#define SALT 3177
#include "CUDA10Kernel.h"
#define SALT 3178
#include "CUDA10Kernel.h"
#define SALT 3179
#include "CUDA10Kernel.h"
#define SALT 3180
#include "CUDA10Kernel.h"
#define SALT 3181
#include "CUDA10Kernel.h"
#define SALT 3182
#include "CUDA10Kernel.h"
#define SALT 3183
#include "CUDA10Kernel.h"
#define SALT 3184
#include "CUDA10Kernel.h"
#define SALT 3185
#include "CUDA10Kernel.h"
#define SALT 3186
#include "CUDA10Kernel.h"
#define SALT 3187
#include "CUDA10Kernel.h"
#define SALT 3188
#include "CUDA10Kernel.h"
#define SALT 3189
#include "CUDA10Kernel.h"
#define SALT 3190
#include "CUDA10Kernel.h"
#define SALT 3191
#include "CUDA10Kernel.h"
#define SALT 3192
#include "CUDA10Kernel.h"
#define SALT 3193
#include "CUDA10Kernel.h"
#define SALT 3194
#include "CUDA10Kernel.h"
#define SALT 3195
#include "CUDA10Kernel.h"
#define SALT 3196
#include "CUDA10Kernel.h"
#define SALT 3197
#include "CUDA10Kernel.h"
#define SALT 3198
#include "CUDA10Kernel.h"
#define SALT 3199
#include "CUDA10Kernel.h"
#define SALT 3200
#include "CUDA10Kernel.h"
#define SALT 3201
#include "CUDA10Kernel.h"
#define SALT 3202
#include "CUDA10Kernel.h"
#define SALT 3203
#include "CUDA10Kernel.h"
#define SALT 3204
#include "CUDA10Kernel.h"
#define SALT 3205
#include "CUDA10Kernel.h"
#define SALT 3206
#include "CUDA10Kernel.h"
#define SALT 3207
#include "CUDA10Kernel.h"
#define SALT 3208
#include "CUDA10Kernel.h"
#define SALT 3209
#include "CUDA10Kernel.h"
#define SALT 3210
#include "CUDA10Kernel.h"
#define SALT 3211
#include "CUDA10Kernel.h"
#define SALT 3212
#include "CUDA10Kernel.h"
#define SALT 3213
#include "CUDA10Kernel.h"
#define SALT 3214
#include "CUDA10Kernel.h"
#define SALT 3215
#include "CUDA10Kernel.h"
#define SALT 3216
#include "CUDA10Kernel.h"
#define SALT 3217
#include "CUDA10Kernel.h"
#define SALT 3218
#include "CUDA10Kernel.h"
#define SALT 3219
#include "CUDA10Kernel.h"
#define SALT 3220
#include "CUDA10Kernel.h"
#define SALT 3221
#include "CUDA10Kernel.h"
#define SALT 3222
#include "CUDA10Kernel.h"
#define SALT 3223
#include "CUDA10Kernel.h"
#define SALT 3224
#include "CUDA10Kernel.h"
#define SALT 3225
#include "CUDA10Kernel.h"
#define SALT 3226
#include "CUDA10Kernel.h"
#define SALT 3227
#include "CUDA10Kernel.h"
#define SALT 3228
#include "CUDA10Kernel.h"
#define SALT 3229
#include "CUDA10Kernel.h"
#define SALT 3230
#include "CUDA10Kernel.h"
#define SALT 3231
#include "CUDA10Kernel.h"
#define SALT 3232
#include "CUDA10Kernel.h"
#define SALT 3233
#include "CUDA10Kernel.h"
#define SALT 3234
#include "CUDA10Kernel.h"
#define SALT 3235
#include "CUDA10Kernel.h"
#define SALT 3236
#include "CUDA10Kernel.h"
#define SALT 3237
#include "CUDA10Kernel.h"
#define SALT 3238
#include "CUDA10Kernel.h"
#define SALT 3239
#include "CUDA10Kernel.h"
#define SALT 3240
#include "CUDA10Kernel.h"
#define SALT 3241
#include "CUDA10Kernel.h"
#define SALT 3242
#include "CUDA10Kernel.h"
#define SALT 3243
#include "CUDA10Kernel.h"
#define SALT 3244
#include "CUDA10Kernel.h"
#define SALT 3245
#include "CUDA10Kernel.h"
#define SALT 3246
#include "CUDA10Kernel.h"
#define SALT 3247
#include "CUDA10Kernel.h"
#define SALT 3248
#include "CUDA10Kernel.h"
#define SALT 3249
#include "CUDA10Kernel.h"
#define SALT 3250
#include "CUDA10Kernel.h"
#define SALT 3251
#include "CUDA10Kernel.h"
#define SALT 3252
#include "CUDA10Kernel.h"
#define SALT 3253
#include "CUDA10Kernel.h"
#define SALT 3254
#include "CUDA10Kernel.h"
#define SALT 3255
#include "CUDA10Kernel.h"
#define SALT 3256
#include "CUDA10Kernel.h"
#define SALT 3257
#include "CUDA10Kernel.h"
#define SALT 3258
#include "CUDA10Kernel.h"
#define SALT 3259
#include "CUDA10Kernel.h"
#define SALT 3260
#include "CUDA10Kernel.h"
#define SALT 3261
#include "CUDA10Kernel.h"
#define SALT 3262
#include "CUDA10Kernel.h"
#define SALT 3263
#include "CUDA10Kernel.h"
#define SALT 3264
#include "CUDA10Kernel.h"
#define SALT 3265
#include "CUDA10Kernel.h"
#define SALT 3266
#include "CUDA10Kernel.h"
#define SALT 3267
#include "CUDA10Kernel.h"
#define SALT 3268
#include "CUDA10Kernel.h"
#define SALT 3269
#include "CUDA10Kernel.h"
#define SALT 3270
#include "CUDA10Kernel.h"
#define SALT 3271
#include "CUDA10Kernel.h"
#define SALT 3272
#include "CUDA10Kernel.h"
#define SALT 3273
#include "CUDA10Kernel.h"
#define SALT 3274
#include "CUDA10Kernel.h"
#define SALT 3275
#include "CUDA10Kernel.h"
#define SALT 3276
#include "CUDA10Kernel.h"
#define SALT 3277
#include "CUDA10Kernel.h"
#define SALT 3278
#include "CUDA10Kernel.h"
#define SALT 3279
#include "CUDA10Kernel.h"
#define SALT 3280
#include "CUDA10Kernel.h"
#define SALT 3281
#include "CUDA10Kernel.h"
#define SALT 3282
#include "CUDA10Kernel.h"
#define SALT 3283
#include "CUDA10Kernel.h"
#define SALT 3284
#include "CUDA10Kernel.h"
#define SALT 3285
#include "CUDA10Kernel.h"
#define SALT 3286
#include "CUDA10Kernel.h"
#define SALT 3287
#include "CUDA10Kernel.h"
#define SALT 3288
#include "CUDA10Kernel.h"
#define SALT 3289
#include "CUDA10Kernel.h"
#define SALT 3290
#include "CUDA10Kernel.h"
#define SALT 3291
#include "CUDA10Kernel.h"
#define SALT 3292
#include "CUDA10Kernel.h"
#define SALT 3293
#include "CUDA10Kernel.h"
#define SALT 3294
#include "CUDA10Kernel.h"
#define SALT 3295
#include "CUDA10Kernel.h"
#define SALT 3296
#include "CUDA10Kernel.h"
#define SALT 3297
#include "CUDA10Kernel.h"
#define SALT 3298
#include "CUDA10Kernel.h"
#define SALT 3299
#include "CUDA10Kernel.h"
#define SALT 3300
#include "CUDA10Kernel.h"
#define SALT 3301
#include "CUDA10Kernel.h"
#define SALT 3302
#include "CUDA10Kernel.h"
#define SALT 3303
#include "CUDA10Kernel.h"
#define SALT 3304
#include "CUDA10Kernel.h"
#define SALT 3305
#include "CUDA10Kernel.h"
#define SALT 3306
#include "CUDA10Kernel.h"
#define SALT 3307
#include "CUDA10Kernel.h"
#define SALT 3308
#include "CUDA10Kernel.h"
#define SALT 3309
#include "CUDA10Kernel.h"
#define SALT 3310
#include "CUDA10Kernel.h"
#define SALT 3311
#include "CUDA10Kernel.h"
#define SALT 3312
#include "CUDA10Kernel.h"
#define SALT 3313
#include "CUDA10Kernel.h"
#define SALT 3314
#include "CUDA10Kernel.h"
#define SALT 3315
#include "CUDA10Kernel.h"
#define SALT 3316
#include "CUDA10Kernel.h"
#define SALT 3317
#include "CUDA10Kernel.h"
#define SALT 3318
#include "CUDA10Kernel.h"
#define SALT 3319
#include "CUDA10Kernel.h"
#define SALT 3320
#include "CUDA10Kernel.h"
#define SALT 3321
#include "CUDA10Kernel.h"
#define SALT 3322
#include "CUDA10Kernel.h"
#define SALT 3323
#include "CUDA10Kernel.h"
#define SALT 3324
#include "CUDA10Kernel.h"
#define SALT 3325
#include "CUDA10Kernel.h"
#define SALT 3326
#include "CUDA10Kernel.h"
#define SALT 3327
#include "CUDA10Kernel.h"
#define SALT 3328
#include "CUDA10Kernel.h"
#define SALT 3329
#include "CUDA10Kernel.h"
#define SALT 3330
#include "CUDA10Kernel.h"
#define SALT 3331
#include "CUDA10Kernel.h"
#define SALT 3332
#include "CUDA10Kernel.h"
#define SALT 3333
#include "CUDA10Kernel.h"
#define SALT 3334
#include "CUDA10Kernel.h"
#define SALT 3335
#include "CUDA10Kernel.h"
#define SALT 3336
#include "CUDA10Kernel.h"
#define SALT 3337
#include "CUDA10Kernel.h"
#define SALT 3338
#include "CUDA10Kernel.h"
#define SALT 3339
#include "CUDA10Kernel.h"
#define SALT 3340
#include "CUDA10Kernel.h"
#define SALT 3341
#include "CUDA10Kernel.h"
#define SALT 3342
#include "CUDA10Kernel.h"
#define SALT 3343
#include "CUDA10Kernel.h"
#define SALT 3344
#include "CUDA10Kernel.h"
#define SALT 3345
#include "CUDA10Kernel.h"
#define SALT 3346
#include "CUDA10Kernel.h"
#define SALT 3347
#include "CUDA10Kernel.h"
#define SALT 3348
#include "CUDA10Kernel.h"
#define SALT 3349
#include "CUDA10Kernel.h"
#define SALT 3350
#include "CUDA10Kernel.h"
#define SALT 3351
#include "CUDA10Kernel.h"
#define SALT 3352
#include "CUDA10Kernel.h"
#define SALT 3353
#include "CUDA10Kernel.h"
#define SALT 3354
#include "CUDA10Kernel.h"
#define SALT 3355
#include "CUDA10Kernel.h"
#define SALT 3356
#include "CUDA10Kernel.h"
#define SALT 3357
#include "CUDA10Kernel.h"
#define SALT 3358
#include "CUDA10Kernel.h"
#define SALT 3359
#include "CUDA10Kernel.h"
#define SALT 3360
#include "CUDA10Kernel.h"
#define SALT 3361
#include "CUDA10Kernel.h"
#define SALT 3362
#include "CUDA10Kernel.h"
#define SALT 3363
#include "CUDA10Kernel.h"
#define SALT 3364
#include "CUDA10Kernel.h"
#define SALT 3365
#include "CUDA10Kernel.h"
#define SALT 3366
#include "CUDA10Kernel.h"
#define SALT 3367
#include "CUDA10Kernel.h"
#define SALT 3368
#include "CUDA10Kernel.h"
#define SALT 3369
#include "CUDA10Kernel.h"
#define SALT 3370
#include "CUDA10Kernel.h"
#define SALT 3371
#include "CUDA10Kernel.h"
#define SALT 3372
#include "CUDA10Kernel.h"
#define SALT 3373
#include "CUDA10Kernel.h"
#define SALT 3374
#include "CUDA10Kernel.h"
#define SALT 3375
#include "CUDA10Kernel.h"
#define SALT 3376
#include "CUDA10Kernel.h"
#define SALT 3377
#include "CUDA10Kernel.h"
#define SALT 3378
#include "CUDA10Kernel.h"
#define SALT 3379
#include "CUDA10Kernel.h"
#define SALT 3380
#include "CUDA10Kernel.h"
#define SALT 3381
#include "CUDA10Kernel.h"
#define SALT 3382
#include "CUDA10Kernel.h"
#define SALT 3383
#include "CUDA10Kernel.h"
#define SALT 3384
#include "CUDA10Kernel.h"
#define SALT 3385
#include "CUDA10Kernel.h"
#define SALT 3386
#include "CUDA10Kernel.h"
#define SALT 3387
#include "CUDA10Kernel.h"
#define SALT 3388
#include "CUDA10Kernel.h"
#define SALT 3389
#include "CUDA10Kernel.h"
#define SALT 3390
#include "CUDA10Kernel.h"
#define SALT 3391
#include "CUDA10Kernel.h"
#define SALT 3392
#include "CUDA10Kernel.h"
#define SALT 3393
#include "CUDA10Kernel.h"
#define SALT 3394
#include "CUDA10Kernel.h"
#define SALT 3395
#include "CUDA10Kernel.h"
#define SALT 3396
#include "CUDA10Kernel.h"
#define SALT 3397
#include "CUDA10Kernel.h"
#define SALT 3398
#include "CUDA10Kernel.h"
#define SALT 3399
#include "CUDA10Kernel.h"
#define SALT 3400
#include "CUDA10Kernel.h"
#define SALT 3401
#include "CUDA10Kernel.h"
#define SALT 3402
#include "CUDA10Kernel.h"
#define SALT 3403
#include "CUDA10Kernel.h"
#define SALT 3404
#include "CUDA10Kernel.h"
#define SALT 3405
#include "CUDA10Kernel.h"
#define SALT 3406
#include "CUDA10Kernel.h"
#define SALT 3407
#include "CUDA10Kernel.h"
#define SALT 3408
#include "CUDA10Kernel.h"
#define SALT 3409
#include "CUDA10Kernel.h"
#define SALT 3410
#include "CUDA10Kernel.h"
#define SALT 3411
#include "CUDA10Kernel.h"
#define SALT 3412
#include "CUDA10Kernel.h"
#define SALT 3413
#include "CUDA10Kernel.h"
#define SALT 3414
#include "CUDA10Kernel.h"
#define SALT 3415
#include "CUDA10Kernel.h"
#define SALT 3416
#include "CUDA10Kernel.h"
#define SALT 3417
#include "CUDA10Kernel.h"
#define SALT 3418
#include "CUDA10Kernel.h"
#define SALT 3419
#include "CUDA10Kernel.h"
#define SALT 3420
#include "CUDA10Kernel.h"
#define SALT 3421
#include "CUDA10Kernel.h"
#define SALT 3422
#include "CUDA10Kernel.h"
#define SALT 3423
#include "CUDA10Kernel.h"
#define SALT 3424
#include "CUDA10Kernel.h"
#define SALT 3425
#include "CUDA10Kernel.h"
#define SALT 3426
#include "CUDA10Kernel.h"
#define SALT 3427
#include "CUDA10Kernel.h"
#define SALT 3428
#include "CUDA10Kernel.h"
#define SALT 3429
#include "CUDA10Kernel.h"
#define SALT 3430
#include "CUDA10Kernel.h"
#define SALT 3431
#include "CUDA10Kernel.h"
#define SALT 3432
#include "CUDA10Kernel.h"
#define SALT 3433
#include "CUDA10Kernel.h"
#define SALT 3434
#include "CUDA10Kernel.h"
#define SALT 3435
#include "CUDA10Kernel.h"
#define SALT 3436
#include "CUDA10Kernel.h"
#define SALT 3437
#include "CUDA10Kernel.h"
#define SALT 3438
#include "CUDA10Kernel.h"
#define SALT 3439
#include "CUDA10Kernel.h"
#define SALT 3440
#include "CUDA10Kernel.h"
#define SALT 3441
#include "CUDA10Kernel.h"
#define SALT 3442
#include "CUDA10Kernel.h"
#define SALT 3443
#include "CUDA10Kernel.h"
#define SALT 3444
#include "CUDA10Kernel.h"
#define SALT 3445
#include "CUDA10Kernel.h"
#define SALT 3446
#include "CUDA10Kernel.h"
#define SALT 3447
#include "CUDA10Kernel.h"
#define SALT 3448
#include "CUDA10Kernel.h"
#define SALT 3449
#include "CUDA10Kernel.h"
#define SALT 3450
#include "CUDA10Kernel.h"
#define SALT 3451
#include "CUDA10Kernel.h"
#define SALT 3452
#include "CUDA10Kernel.h"
#define SALT 3453
#include "CUDA10Kernel.h"
#define SALT 3454
#include "CUDA10Kernel.h"
#define SALT 3455
#include "CUDA10Kernel.h"
#define SALT 3456
#include "CUDA10Kernel.h"
#define SALT 3457
#include "CUDA10Kernel.h"
#define SALT 3458
#include "CUDA10Kernel.h"
#define SALT 3459
#include "CUDA10Kernel.h"
#define SALT 3460
#include "CUDA10Kernel.h"
#define SALT 3461
#include "CUDA10Kernel.h"
#define SALT 3462
#include "CUDA10Kernel.h"
#define SALT 3463
#include "CUDA10Kernel.h"
#define SALT 3464
#include "CUDA10Kernel.h"
#define SALT 3465
#include "CUDA10Kernel.h"
#define SALT 3466
#include "CUDA10Kernel.h"
#define SALT 3467
#include "CUDA10Kernel.h"
#define SALT 3468
#include "CUDA10Kernel.h"
#define SALT 3469
#include "CUDA10Kernel.h"
#define SALT 3470
#include "CUDA10Kernel.h"
#define SALT 3471
#include "CUDA10Kernel.h"
#define SALT 3472
#include "CUDA10Kernel.h"
#define SALT 3473
#include "CUDA10Kernel.h"
#define SALT 3474
#include "CUDA10Kernel.h"
#define SALT 3475
#include "CUDA10Kernel.h"
#define SALT 3476
#include "CUDA10Kernel.h"
#define SALT 3477
#include "CUDA10Kernel.h"
#define SALT 3478
#include "CUDA10Kernel.h"
#define SALT 3479
#include "CUDA10Kernel.h"
#define SALT 3480
#include "CUDA10Kernel.h"
#define SALT 3481
#include "CUDA10Kernel.h"
#define SALT 3482
#include "CUDA10Kernel.h"
#define SALT 3483
#include "CUDA10Kernel.h"
#define SALT 3484
#include "CUDA10Kernel.h"
#define SALT 3485
#include "CUDA10Kernel.h"
#define SALT 3486
#include "CUDA10Kernel.h"
#define SALT 3487
#include "CUDA10Kernel.h"
#define SALT 3488
#include "CUDA10Kernel.h"
#define SALT 3489
#include "CUDA10Kernel.h"
#define SALT 3490
#include "CUDA10Kernel.h"
#define SALT 3491
#include "CUDA10Kernel.h"
#define SALT 3492
#include "CUDA10Kernel.h"
#define SALT 3493
#include "CUDA10Kernel.h"
#define SALT 3494
#include "CUDA10Kernel.h"
#define SALT 3495
#include "CUDA10Kernel.h"
#define SALT 3496
#include "CUDA10Kernel.h"
#define SALT 3497
#include "CUDA10Kernel.h"
#define SALT 3498
#include "CUDA10Kernel.h"
#define SALT 3499
#include "CUDA10Kernel.h"
#define SALT 3500
#include "CUDA10Kernel.h"
#define SALT 3501
#include "CUDA10Kernel.h"
#define SALT 3502
#include "CUDA10Kernel.h"
#define SALT 3503
#include "CUDA10Kernel.h"
#define SALT 3504
#include "CUDA10Kernel.h"
#define SALT 3505
#include "CUDA10Kernel.h"
#define SALT 3506
#include "CUDA10Kernel.h"
#define SALT 3507
#include "CUDA10Kernel.h"
#define SALT 3508
#include "CUDA10Kernel.h"
#define SALT 3509
#include "CUDA10Kernel.h"
#define SALT 3510
#include "CUDA10Kernel.h"
#define SALT 3511
#include "CUDA10Kernel.h"
#define SALT 3512
#include "CUDA10Kernel.h"
#define SALT 3513
#include "CUDA10Kernel.h"
#define SALT 3514
#include "CUDA10Kernel.h"
#define SALT 3515
#include "CUDA10Kernel.h"
#define SALT 3516
#include "CUDA10Kernel.h"
#define SALT 3517
#include "CUDA10Kernel.h"
#define SALT 3518
#include "CUDA10Kernel.h"
#define SALT 3519
#include "CUDA10Kernel.h"
#define SALT 3520
#include "CUDA10Kernel.h"
#define SALT 3521
#include "CUDA10Kernel.h"
#define SALT 3522
#include "CUDA10Kernel.h"
#define SALT 3523
#include "CUDA10Kernel.h"
#define SALT 3524
#include "CUDA10Kernel.h"
#define SALT 3525
#include "CUDA10Kernel.h"
#define SALT 3526
#include "CUDA10Kernel.h"
#define SALT 3527
#include "CUDA10Kernel.h"
#define SALT 3528
#include "CUDA10Kernel.h"
#define SALT 3529
#include "CUDA10Kernel.h"
#define SALT 3530
#include "CUDA10Kernel.h"
#define SALT 3531
#include "CUDA10Kernel.h"
#define SALT 3532
#include "CUDA10Kernel.h"
#define SALT 3533
#include "CUDA10Kernel.h"
#define SALT 3534
#include "CUDA10Kernel.h"
#define SALT 3535
#include "CUDA10Kernel.h"
#define SALT 3536
#include "CUDA10Kernel.h"
#define SALT 3537
#include "CUDA10Kernel.h"
#define SALT 3538
#include "CUDA10Kernel.h"
#define SALT 3539
#include "CUDA10Kernel.h"
#define SALT 3540
#include "CUDA10Kernel.h"
#define SALT 3541
#include "CUDA10Kernel.h"
#define SALT 3542
#include "CUDA10Kernel.h"
#define SALT 3543
#include "CUDA10Kernel.h"
#define SALT 3544
#include "CUDA10Kernel.h"
#define SALT 3545
#include "CUDA10Kernel.h"
#define SALT 3546
#include "CUDA10Kernel.h"
#define SALT 3547
#include "CUDA10Kernel.h"
#define SALT 3548
#include "CUDA10Kernel.h"
#define SALT 3549
#include "CUDA10Kernel.h"
#define SALT 3550
#include "CUDA10Kernel.h"
#define SALT 3551
#include "CUDA10Kernel.h"
#define SALT 3552
#include "CUDA10Kernel.h"
#define SALT 3553
#include "CUDA10Kernel.h"
#define SALT 3554
#include "CUDA10Kernel.h"
#define SALT 3555
#include "CUDA10Kernel.h"
#define SALT 3556
#include "CUDA10Kernel.h"
#define SALT 3557
#include "CUDA10Kernel.h"
#define SALT 3558
#include "CUDA10Kernel.h"
#define SALT 3559
#include "CUDA10Kernel.h"
#define SALT 3560
#include "CUDA10Kernel.h"
#define SALT 3561
#include "CUDA10Kernel.h"
#define SALT 3562
#include "CUDA10Kernel.h"
#define SALT 3563
#include "CUDA10Kernel.h"
#define SALT 3564
#include "CUDA10Kernel.h"
#define SALT 3565
#include "CUDA10Kernel.h"
#define SALT 3566
#include "CUDA10Kernel.h"
#define SALT 3567
#include "CUDA10Kernel.h"
#define SALT 3568
#include "CUDA10Kernel.h"
#define SALT 3569
#include "CUDA10Kernel.h"
#define SALT 3570
#include "CUDA10Kernel.h"
#define SALT 3571
#include "CUDA10Kernel.h"
#define SALT 3572
#include "CUDA10Kernel.h"
#define SALT 3573
#include "CUDA10Kernel.h"
#define SALT 3574
#include "CUDA10Kernel.h"
#define SALT 3575
#include "CUDA10Kernel.h"
#define SALT 3576
#include "CUDA10Kernel.h"
#define SALT 3577
#include "CUDA10Kernel.h"
#define SALT 3578
#include "CUDA10Kernel.h"
#define SALT 3579
#include "CUDA10Kernel.h"
#define SALT 3580
#include "CUDA10Kernel.h"
#define SALT 3581
#include "CUDA10Kernel.h"
#define SALT 3582
#include "CUDA10Kernel.h"
#define SALT 3583
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher6()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel6(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 3072: CUDA_DES_PerformSearch3072<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3073: CUDA_DES_PerformSearch3073<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3074: CUDA_DES_PerformSearch3074<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3075: CUDA_DES_PerformSearch3075<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3076: CUDA_DES_PerformSearch3076<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3077: CUDA_DES_PerformSearch3077<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3078: CUDA_DES_PerformSearch3078<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3079: CUDA_DES_PerformSearch3079<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3080: CUDA_DES_PerformSearch3080<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3081: CUDA_DES_PerformSearch3081<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3082: CUDA_DES_PerformSearch3082<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3083: CUDA_DES_PerformSearch3083<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3084: CUDA_DES_PerformSearch3084<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3085: CUDA_DES_PerformSearch3085<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3086: CUDA_DES_PerformSearch3086<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3087: CUDA_DES_PerformSearch3087<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3088: CUDA_DES_PerformSearch3088<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3089: CUDA_DES_PerformSearch3089<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3090: CUDA_DES_PerformSearch3090<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3091: CUDA_DES_PerformSearch3091<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3092: CUDA_DES_PerformSearch3092<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3093: CUDA_DES_PerformSearch3093<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3094: CUDA_DES_PerformSearch3094<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3095: CUDA_DES_PerformSearch3095<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3096: CUDA_DES_PerformSearch3096<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3097: CUDA_DES_PerformSearch3097<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3098: CUDA_DES_PerformSearch3098<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3099: CUDA_DES_PerformSearch3099<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3100: CUDA_DES_PerformSearch3100<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3101: CUDA_DES_PerformSearch3101<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3102: CUDA_DES_PerformSearch3102<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3103: CUDA_DES_PerformSearch3103<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3104: CUDA_DES_PerformSearch3104<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3105: CUDA_DES_PerformSearch3105<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3106: CUDA_DES_PerformSearch3106<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3107: CUDA_DES_PerformSearch3107<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3108: CUDA_DES_PerformSearch3108<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3109: CUDA_DES_PerformSearch3109<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3110: CUDA_DES_PerformSearch3110<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3111: CUDA_DES_PerformSearch3111<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3112: CUDA_DES_PerformSearch3112<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3113: CUDA_DES_PerformSearch3113<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3114: CUDA_DES_PerformSearch3114<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3115: CUDA_DES_PerformSearch3115<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3116: CUDA_DES_PerformSearch3116<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3117: CUDA_DES_PerformSearch3117<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3118: CUDA_DES_PerformSearch3118<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3119: CUDA_DES_PerformSearch3119<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3120: CUDA_DES_PerformSearch3120<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3121: CUDA_DES_PerformSearch3121<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3122: CUDA_DES_PerformSearch3122<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3123: CUDA_DES_PerformSearch3123<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3124: CUDA_DES_PerformSearch3124<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3125: CUDA_DES_PerformSearch3125<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3126: CUDA_DES_PerformSearch3126<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3127: CUDA_DES_PerformSearch3127<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3128: CUDA_DES_PerformSearch3128<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3129: CUDA_DES_PerformSearch3129<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3130: CUDA_DES_PerformSearch3130<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3131: CUDA_DES_PerformSearch3131<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3132: CUDA_DES_PerformSearch3132<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3133: CUDA_DES_PerformSearch3133<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3134: CUDA_DES_PerformSearch3134<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3135: CUDA_DES_PerformSearch3135<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3136: CUDA_DES_PerformSearch3136<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3137: CUDA_DES_PerformSearch3137<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3138: CUDA_DES_PerformSearch3138<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3139: CUDA_DES_PerformSearch3139<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3140: CUDA_DES_PerformSearch3140<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3141: CUDA_DES_PerformSearch3141<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3142: CUDA_DES_PerformSearch3142<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3143: CUDA_DES_PerformSearch3143<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3144: CUDA_DES_PerformSearch3144<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3145: CUDA_DES_PerformSearch3145<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3146: CUDA_DES_PerformSearch3146<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3147: CUDA_DES_PerformSearch3147<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3148: CUDA_DES_PerformSearch3148<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3149: CUDA_DES_PerformSearch3149<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3150: CUDA_DES_PerformSearch3150<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3151: CUDA_DES_PerformSearch3151<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3152: CUDA_DES_PerformSearch3152<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3153: CUDA_DES_PerformSearch3153<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3154: CUDA_DES_PerformSearch3154<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3155: CUDA_DES_PerformSearch3155<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3156: CUDA_DES_PerformSearch3156<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3157: CUDA_DES_PerformSearch3157<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3158: CUDA_DES_PerformSearch3158<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3159: CUDA_DES_PerformSearch3159<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3160: CUDA_DES_PerformSearch3160<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3161: CUDA_DES_PerformSearch3161<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3162: CUDA_DES_PerformSearch3162<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3163: CUDA_DES_PerformSearch3163<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3164: CUDA_DES_PerformSearch3164<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3165: CUDA_DES_PerformSearch3165<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3166: CUDA_DES_PerformSearch3166<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3167: CUDA_DES_PerformSearch3167<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3168: CUDA_DES_PerformSearch3168<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3169: CUDA_DES_PerformSearch3169<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3170: CUDA_DES_PerformSearch3170<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3171: CUDA_DES_PerformSearch3171<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3172: CUDA_DES_PerformSearch3172<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3173: CUDA_DES_PerformSearch3173<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3174: CUDA_DES_PerformSearch3174<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3175: CUDA_DES_PerformSearch3175<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3176: CUDA_DES_PerformSearch3176<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3177: CUDA_DES_PerformSearch3177<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3178: CUDA_DES_PerformSearch3178<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3179: CUDA_DES_PerformSearch3179<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3180: CUDA_DES_PerformSearch3180<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3181: CUDA_DES_PerformSearch3181<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3182: CUDA_DES_PerformSearch3182<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3183: CUDA_DES_PerformSearch3183<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3184: CUDA_DES_PerformSearch3184<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3185: CUDA_DES_PerformSearch3185<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3186: CUDA_DES_PerformSearch3186<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3187: CUDA_DES_PerformSearch3187<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3188: CUDA_DES_PerformSearch3188<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3189: CUDA_DES_PerformSearch3189<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3190: CUDA_DES_PerformSearch3190<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3191: CUDA_DES_PerformSearch3191<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3192: CUDA_DES_PerformSearch3192<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3193: CUDA_DES_PerformSearch3193<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3194: CUDA_DES_PerformSearch3194<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3195: CUDA_DES_PerformSearch3195<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3196: CUDA_DES_PerformSearch3196<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3197: CUDA_DES_PerformSearch3197<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3198: CUDA_DES_PerformSearch3198<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3199: CUDA_DES_PerformSearch3199<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3200: CUDA_DES_PerformSearch3200<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3201: CUDA_DES_PerformSearch3201<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3202: CUDA_DES_PerformSearch3202<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3203: CUDA_DES_PerformSearch3203<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3204: CUDA_DES_PerformSearch3204<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3205: CUDA_DES_PerformSearch3205<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3206: CUDA_DES_PerformSearch3206<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3207: CUDA_DES_PerformSearch3207<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3208: CUDA_DES_PerformSearch3208<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3209: CUDA_DES_PerformSearch3209<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3210: CUDA_DES_PerformSearch3210<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3211: CUDA_DES_PerformSearch3211<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3212: CUDA_DES_PerformSearch3212<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3213: CUDA_DES_PerformSearch3213<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3214: CUDA_DES_PerformSearch3214<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3215: CUDA_DES_PerformSearch3215<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3216: CUDA_DES_PerformSearch3216<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3217: CUDA_DES_PerformSearch3217<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3218: CUDA_DES_PerformSearch3218<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3219: CUDA_DES_PerformSearch3219<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3220: CUDA_DES_PerformSearch3220<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3221: CUDA_DES_PerformSearch3221<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3222: CUDA_DES_PerformSearch3222<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3223: CUDA_DES_PerformSearch3223<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3224: CUDA_DES_PerformSearch3224<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3225: CUDA_DES_PerformSearch3225<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3226: CUDA_DES_PerformSearch3226<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3227: CUDA_DES_PerformSearch3227<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3228: CUDA_DES_PerformSearch3228<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3229: CUDA_DES_PerformSearch3229<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3230: CUDA_DES_PerformSearch3230<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3231: CUDA_DES_PerformSearch3231<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3232: CUDA_DES_PerformSearch3232<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3233: CUDA_DES_PerformSearch3233<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3234: CUDA_DES_PerformSearch3234<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3235: CUDA_DES_PerformSearch3235<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3236: CUDA_DES_PerformSearch3236<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3237: CUDA_DES_PerformSearch3237<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3238: CUDA_DES_PerformSearch3238<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3239: CUDA_DES_PerformSearch3239<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3240: CUDA_DES_PerformSearch3240<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3241: CUDA_DES_PerformSearch3241<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3242: CUDA_DES_PerformSearch3242<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3243: CUDA_DES_PerformSearch3243<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3244: CUDA_DES_PerformSearch3244<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3245: CUDA_DES_PerformSearch3245<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3246: CUDA_DES_PerformSearch3246<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3247: CUDA_DES_PerformSearch3247<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3248: CUDA_DES_PerformSearch3248<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3249: CUDA_DES_PerformSearch3249<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3250: CUDA_DES_PerformSearch3250<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3251: CUDA_DES_PerformSearch3251<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3252: CUDA_DES_PerformSearch3252<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3253: CUDA_DES_PerformSearch3253<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3254: CUDA_DES_PerformSearch3254<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3255: CUDA_DES_PerformSearch3255<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3256: CUDA_DES_PerformSearch3256<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3257: CUDA_DES_PerformSearch3257<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3258: CUDA_DES_PerformSearch3258<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3259: CUDA_DES_PerformSearch3259<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3260: CUDA_DES_PerformSearch3260<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3261: CUDA_DES_PerformSearch3261<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3262: CUDA_DES_PerformSearch3262<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3263: CUDA_DES_PerformSearch3263<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3264: CUDA_DES_PerformSearch3264<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3265: CUDA_DES_PerformSearch3265<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3266: CUDA_DES_PerformSearch3266<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3267: CUDA_DES_PerformSearch3267<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3268: CUDA_DES_PerformSearch3268<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3269: CUDA_DES_PerformSearch3269<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3270: CUDA_DES_PerformSearch3270<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3271: CUDA_DES_PerformSearch3271<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3272: CUDA_DES_PerformSearch3272<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3273: CUDA_DES_PerformSearch3273<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3274: CUDA_DES_PerformSearch3274<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3275: CUDA_DES_PerformSearch3275<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3276: CUDA_DES_PerformSearch3276<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3277: CUDA_DES_PerformSearch3277<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3278: CUDA_DES_PerformSearch3278<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3279: CUDA_DES_PerformSearch3279<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3280: CUDA_DES_PerformSearch3280<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3281: CUDA_DES_PerformSearch3281<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3282: CUDA_DES_PerformSearch3282<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3283: CUDA_DES_PerformSearch3283<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3284: CUDA_DES_PerformSearch3284<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3285: CUDA_DES_PerformSearch3285<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3286: CUDA_DES_PerformSearch3286<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3287: CUDA_DES_PerformSearch3287<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3288: CUDA_DES_PerformSearch3288<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3289: CUDA_DES_PerformSearch3289<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3290: CUDA_DES_PerformSearch3290<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3291: CUDA_DES_PerformSearch3291<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3292: CUDA_DES_PerformSearch3292<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3293: CUDA_DES_PerformSearch3293<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3294: CUDA_DES_PerformSearch3294<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3295: CUDA_DES_PerformSearch3295<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3296: CUDA_DES_PerformSearch3296<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3297: CUDA_DES_PerformSearch3297<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3298: CUDA_DES_PerformSearch3298<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3299: CUDA_DES_PerformSearch3299<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3300: CUDA_DES_PerformSearch3300<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3301: CUDA_DES_PerformSearch3301<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3302: CUDA_DES_PerformSearch3302<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3303: CUDA_DES_PerformSearch3303<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3304: CUDA_DES_PerformSearch3304<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3305: CUDA_DES_PerformSearch3305<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3306: CUDA_DES_PerformSearch3306<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3307: CUDA_DES_PerformSearch3307<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3308: CUDA_DES_PerformSearch3308<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3309: CUDA_DES_PerformSearch3309<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3310: CUDA_DES_PerformSearch3310<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3311: CUDA_DES_PerformSearch3311<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3312: CUDA_DES_PerformSearch3312<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3313: CUDA_DES_PerformSearch3313<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3314: CUDA_DES_PerformSearch3314<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3315: CUDA_DES_PerformSearch3315<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3316: CUDA_DES_PerformSearch3316<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3317: CUDA_DES_PerformSearch3317<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3318: CUDA_DES_PerformSearch3318<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3319: CUDA_DES_PerformSearch3319<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3320: CUDA_DES_PerformSearch3320<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3321: CUDA_DES_PerformSearch3321<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3322: CUDA_DES_PerformSearch3322<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3323: CUDA_DES_PerformSearch3323<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3324: CUDA_DES_PerformSearch3324<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3325: CUDA_DES_PerformSearch3325<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3326: CUDA_DES_PerformSearch3326<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3327: CUDA_DES_PerformSearch3327<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3328: CUDA_DES_PerformSearch3328<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3329: CUDA_DES_PerformSearch3329<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3330: CUDA_DES_PerformSearch3330<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3331: CUDA_DES_PerformSearch3331<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3332: CUDA_DES_PerformSearch3332<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3333: CUDA_DES_PerformSearch3333<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3334: CUDA_DES_PerformSearch3334<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3335: CUDA_DES_PerformSearch3335<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3336: CUDA_DES_PerformSearch3336<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3337: CUDA_DES_PerformSearch3337<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3338: CUDA_DES_PerformSearch3338<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3339: CUDA_DES_PerformSearch3339<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3340: CUDA_DES_PerformSearch3340<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3341: CUDA_DES_PerformSearch3341<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3342: CUDA_DES_PerformSearch3342<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3343: CUDA_DES_PerformSearch3343<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3344: CUDA_DES_PerformSearch3344<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3345: CUDA_DES_PerformSearch3345<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3346: CUDA_DES_PerformSearch3346<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3347: CUDA_DES_PerformSearch3347<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3348: CUDA_DES_PerformSearch3348<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3349: CUDA_DES_PerformSearch3349<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3350: CUDA_DES_PerformSearch3350<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3351: CUDA_DES_PerformSearch3351<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3352: CUDA_DES_PerformSearch3352<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3353: CUDA_DES_PerformSearch3353<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3354: CUDA_DES_PerformSearch3354<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3355: CUDA_DES_PerformSearch3355<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3356: CUDA_DES_PerformSearch3356<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3357: CUDA_DES_PerformSearch3357<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3358: CUDA_DES_PerformSearch3358<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3359: CUDA_DES_PerformSearch3359<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3360: CUDA_DES_PerformSearch3360<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3361: CUDA_DES_PerformSearch3361<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3362: CUDA_DES_PerformSearch3362<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3363: CUDA_DES_PerformSearch3363<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3364: CUDA_DES_PerformSearch3364<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3365: CUDA_DES_PerformSearch3365<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3366: CUDA_DES_PerformSearch3366<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3367: CUDA_DES_PerformSearch3367<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3368: CUDA_DES_PerformSearch3368<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3369: CUDA_DES_PerformSearch3369<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3370: CUDA_DES_PerformSearch3370<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3371: CUDA_DES_PerformSearch3371<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3372: CUDA_DES_PerformSearch3372<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3373: CUDA_DES_PerformSearch3373<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3374: CUDA_DES_PerformSearch3374<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3375: CUDA_DES_PerformSearch3375<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3376: CUDA_DES_PerformSearch3376<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3377: CUDA_DES_PerformSearch3377<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3378: CUDA_DES_PerformSearch3378<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3379: CUDA_DES_PerformSearch3379<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3380: CUDA_DES_PerformSearch3380<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3381: CUDA_DES_PerformSearch3381<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3382: CUDA_DES_PerformSearch3382<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3383: CUDA_DES_PerformSearch3383<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3384: CUDA_DES_PerformSearch3384<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3385: CUDA_DES_PerformSearch3385<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3386: CUDA_DES_PerformSearch3386<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3387: CUDA_DES_PerformSearch3387<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3388: CUDA_DES_PerformSearch3388<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3389: CUDA_DES_PerformSearch3389<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3390: CUDA_DES_PerformSearch3390<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3391: CUDA_DES_PerformSearch3391<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3392: CUDA_DES_PerformSearch3392<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3393: CUDA_DES_PerformSearch3393<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3394: CUDA_DES_PerformSearch3394<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3395: CUDA_DES_PerformSearch3395<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3396: CUDA_DES_PerformSearch3396<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3397: CUDA_DES_PerformSearch3397<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3398: CUDA_DES_PerformSearch3398<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3399: CUDA_DES_PerformSearch3399<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3400: CUDA_DES_PerformSearch3400<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3401: CUDA_DES_PerformSearch3401<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3402: CUDA_DES_PerformSearch3402<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3403: CUDA_DES_PerformSearch3403<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3404: CUDA_DES_PerformSearch3404<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3405: CUDA_DES_PerformSearch3405<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3406: CUDA_DES_PerformSearch3406<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3407: CUDA_DES_PerformSearch3407<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3408: CUDA_DES_PerformSearch3408<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3409: CUDA_DES_PerformSearch3409<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3410: CUDA_DES_PerformSearch3410<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3411: CUDA_DES_PerformSearch3411<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3412: CUDA_DES_PerformSearch3412<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3413: CUDA_DES_PerformSearch3413<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3414: CUDA_DES_PerformSearch3414<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3415: CUDA_DES_PerformSearch3415<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3416: CUDA_DES_PerformSearch3416<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3417: CUDA_DES_PerformSearch3417<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3418: CUDA_DES_PerformSearch3418<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3419: CUDA_DES_PerformSearch3419<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3420: CUDA_DES_PerformSearch3420<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3421: CUDA_DES_PerformSearch3421<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3422: CUDA_DES_PerformSearch3422<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3423: CUDA_DES_PerformSearch3423<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3424: CUDA_DES_PerformSearch3424<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3425: CUDA_DES_PerformSearch3425<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3426: CUDA_DES_PerformSearch3426<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3427: CUDA_DES_PerformSearch3427<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3428: CUDA_DES_PerformSearch3428<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3429: CUDA_DES_PerformSearch3429<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3430: CUDA_DES_PerformSearch3430<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3431: CUDA_DES_PerformSearch3431<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3432: CUDA_DES_PerformSearch3432<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3433: CUDA_DES_PerformSearch3433<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3434: CUDA_DES_PerformSearch3434<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3435: CUDA_DES_PerformSearch3435<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3436: CUDA_DES_PerformSearch3436<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3437: CUDA_DES_PerformSearch3437<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3438: CUDA_DES_PerformSearch3438<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3439: CUDA_DES_PerformSearch3439<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3440: CUDA_DES_PerformSearch3440<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3441: CUDA_DES_PerformSearch3441<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3442: CUDA_DES_PerformSearch3442<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3443: CUDA_DES_PerformSearch3443<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3444: CUDA_DES_PerformSearch3444<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3445: CUDA_DES_PerformSearch3445<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3446: CUDA_DES_PerformSearch3446<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3447: CUDA_DES_PerformSearch3447<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3448: CUDA_DES_PerformSearch3448<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3449: CUDA_DES_PerformSearch3449<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3450: CUDA_DES_PerformSearch3450<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3451: CUDA_DES_PerformSearch3451<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3452: CUDA_DES_PerformSearch3452<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3453: CUDA_DES_PerformSearch3453<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3454: CUDA_DES_PerformSearch3454<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3455: CUDA_DES_PerformSearch3455<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3456: CUDA_DES_PerformSearch3456<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3457: CUDA_DES_PerformSearch3457<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3458: CUDA_DES_PerformSearch3458<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3459: CUDA_DES_PerformSearch3459<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3460: CUDA_DES_PerformSearch3460<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3461: CUDA_DES_PerformSearch3461<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3462: CUDA_DES_PerformSearch3462<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3463: CUDA_DES_PerformSearch3463<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3464: CUDA_DES_PerformSearch3464<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3465: CUDA_DES_PerformSearch3465<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3466: CUDA_DES_PerformSearch3466<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3467: CUDA_DES_PerformSearch3467<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3468: CUDA_DES_PerformSearch3468<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3469: CUDA_DES_PerformSearch3469<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3470: CUDA_DES_PerformSearch3470<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3471: CUDA_DES_PerformSearch3471<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3472: CUDA_DES_PerformSearch3472<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3473: CUDA_DES_PerformSearch3473<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3474: CUDA_DES_PerformSearch3474<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3475: CUDA_DES_PerformSearch3475<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3476: CUDA_DES_PerformSearch3476<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3477: CUDA_DES_PerformSearch3477<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3478: CUDA_DES_PerformSearch3478<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3479: CUDA_DES_PerformSearch3479<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3480: CUDA_DES_PerformSearch3480<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3481: CUDA_DES_PerformSearch3481<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3482: CUDA_DES_PerformSearch3482<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3483: CUDA_DES_PerformSearch3483<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3484: CUDA_DES_PerformSearch3484<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3485: CUDA_DES_PerformSearch3485<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3486: CUDA_DES_PerformSearch3486<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3487: CUDA_DES_PerformSearch3487<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3488: CUDA_DES_PerformSearch3488<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3489: CUDA_DES_PerformSearch3489<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3490: CUDA_DES_PerformSearch3490<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3491: CUDA_DES_PerformSearch3491<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3492: CUDA_DES_PerformSearch3492<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3493: CUDA_DES_PerformSearch3493<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3494: CUDA_DES_PerformSearch3494<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3495: CUDA_DES_PerformSearch3495<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3496: CUDA_DES_PerformSearch3496<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3497: CUDA_DES_PerformSearch3497<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3498: CUDA_DES_PerformSearch3498<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3499: CUDA_DES_PerformSearch3499<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3500: CUDA_DES_PerformSearch3500<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3501: CUDA_DES_PerformSearch3501<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3502: CUDA_DES_PerformSearch3502<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3503: CUDA_DES_PerformSearch3503<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3504: CUDA_DES_PerformSearch3504<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3505: CUDA_DES_PerformSearch3505<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3506: CUDA_DES_PerformSearch3506<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3507: CUDA_DES_PerformSearch3507<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3508: CUDA_DES_PerformSearch3508<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3509: CUDA_DES_PerformSearch3509<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3510: CUDA_DES_PerformSearch3510<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3511: CUDA_DES_PerformSearch3511<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3512: CUDA_DES_PerformSearch3512<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3513: CUDA_DES_PerformSearch3513<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3514: CUDA_DES_PerformSearch3514<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3515: CUDA_DES_PerformSearch3515<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3516: CUDA_DES_PerformSearch3516<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3517: CUDA_DES_PerformSearch3517<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3518: CUDA_DES_PerformSearch3518<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3519: CUDA_DES_PerformSearch3519<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3520: CUDA_DES_PerformSearch3520<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3521: CUDA_DES_PerformSearch3521<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3522: CUDA_DES_PerformSearch3522<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3523: CUDA_DES_PerformSearch3523<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3524: CUDA_DES_PerformSearch3524<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3525: CUDA_DES_PerformSearch3525<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3526: CUDA_DES_PerformSearch3526<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3527: CUDA_DES_PerformSearch3527<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3528: CUDA_DES_PerformSearch3528<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3529: CUDA_DES_PerformSearch3529<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3530: CUDA_DES_PerformSearch3530<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3531: CUDA_DES_PerformSearch3531<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3532: CUDA_DES_PerformSearch3532<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3533: CUDA_DES_PerformSearch3533<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3534: CUDA_DES_PerformSearch3534<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3535: CUDA_DES_PerformSearch3535<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3536: CUDA_DES_PerformSearch3536<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3537: CUDA_DES_PerformSearch3537<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3538: CUDA_DES_PerformSearch3538<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3539: CUDA_DES_PerformSearch3539<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3540: CUDA_DES_PerformSearch3540<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3541: CUDA_DES_PerformSearch3541<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3542: CUDA_DES_PerformSearch3542<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3543: CUDA_DES_PerformSearch3543<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3544: CUDA_DES_PerformSearch3544<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3545: CUDA_DES_PerformSearch3545<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3546: CUDA_DES_PerformSearch3546<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3547: CUDA_DES_PerformSearch3547<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3548: CUDA_DES_PerformSearch3548<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3549: CUDA_DES_PerformSearch3549<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3550: CUDA_DES_PerformSearch3550<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3551: CUDA_DES_PerformSearch3551<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3552: CUDA_DES_PerformSearch3552<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3553: CUDA_DES_PerformSearch3553<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3554: CUDA_DES_PerformSearch3554<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3555: CUDA_DES_PerformSearch3555<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3556: CUDA_DES_PerformSearch3556<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3557: CUDA_DES_PerformSearch3557<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3558: CUDA_DES_PerformSearch3558<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3559: CUDA_DES_PerformSearch3559<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3560: CUDA_DES_PerformSearch3560<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3561: CUDA_DES_PerformSearch3561<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3562: CUDA_DES_PerformSearch3562<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3563: CUDA_DES_PerformSearch3563<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3564: CUDA_DES_PerformSearch3564<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3565: CUDA_DES_PerformSearch3565<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3566: CUDA_DES_PerformSearch3566<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3567: CUDA_DES_PerformSearch3567<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3568: CUDA_DES_PerformSearch3568<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3569: CUDA_DES_PerformSearch3569<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3570: CUDA_DES_PerformSearch3570<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3571: CUDA_DES_PerformSearch3571<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3572: CUDA_DES_PerformSearch3572<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3573: CUDA_DES_PerformSearch3573<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3574: CUDA_DES_PerformSearch3574<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3575: CUDA_DES_PerformSearch3575<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3576: CUDA_DES_PerformSearch3576<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3577: CUDA_DES_PerformSearch3577<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3578: CUDA_DES_PerformSearch3578<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3579: CUDA_DES_PerformSearch3579<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3580: CUDA_DES_PerformSearch3580<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3581: CUDA_DES_PerformSearch3581<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3582: CUDA_DES_PerformSearch3582<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3583: CUDA_DES_PerformSearch3583<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
