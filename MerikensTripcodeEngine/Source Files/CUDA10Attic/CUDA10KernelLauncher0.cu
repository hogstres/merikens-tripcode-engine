// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 0
#include "CUDA10Kernel.h"
#define SALT 1
#include "CUDA10Kernel.h"
#define SALT 2
#include "CUDA10Kernel.h"
#define SALT 3
#include "CUDA10Kernel.h"
#define SALT 4
#include "CUDA10Kernel.h"
#define SALT 5
#include "CUDA10Kernel.h"
#define SALT 6
#include "CUDA10Kernel.h"
#define SALT 7
#include "CUDA10Kernel.h"
#define SALT 8
#include "CUDA10Kernel.h"
#define SALT 9
#include "CUDA10Kernel.h"
#define SALT 10
#include "CUDA10Kernel.h"
#define SALT 11
#include "CUDA10Kernel.h"
#define SALT 12
#include "CUDA10Kernel.h"
#define SALT 13
#include "CUDA10Kernel.h"
#define SALT 14
#include "CUDA10Kernel.h"
#define SALT 15
#include "CUDA10Kernel.h"
#define SALT 16
#include "CUDA10Kernel.h"
#define SALT 17
#include "CUDA10Kernel.h"
#define SALT 18
#include "CUDA10Kernel.h"
#define SALT 19
#include "CUDA10Kernel.h"
#define SALT 20
#include "CUDA10Kernel.h"
#define SALT 21
#include "CUDA10Kernel.h"
#define SALT 22
#include "CUDA10Kernel.h"
#define SALT 23
#include "CUDA10Kernel.h"
#define SALT 24
#include "CUDA10Kernel.h"
#define SALT 25
#include "CUDA10Kernel.h"
#define SALT 26
#include "CUDA10Kernel.h"
#define SALT 27
#include "CUDA10Kernel.h"
#define SALT 28
#include "CUDA10Kernel.h"
#define SALT 29
#include "CUDA10Kernel.h"
#define SALT 30
#include "CUDA10Kernel.h"
#define SALT 31
#include "CUDA10Kernel.h"
#define SALT 32
#include "CUDA10Kernel.h"
#define SALT 33
#include "CUDA10Kernel.h"
#define SALT 34
#include "CUDA10Kernel.h"
#define SALT 35
#include "CUDA10Kernel.h"
#define SALT 36
#include "CUDA10Kernel.h"
#define SALT 37
#include "CUDA10Kernel.h"
#define SALT 38
#include "CUDA10Kernel.h"
#define SALT 39
#include "CUDA10Kernel.h"
#define SALT 40
#include "CUDA10Kernel.h"
#define SALT 41
#include "CUDA10Kernel.h"
#define SALT 42
#include "CUDA10Kernel.h"
#define SALT 43
#include "CUDA10Kernel.h"
#define SALT 44
#include "CUDA10Kernel.h"
#define SALT 45
#include "CUDA10Kernel.h"
#define SALT 46
#include "CUDA10Kernel.h"
#define SALT 47
#include "CUDA10Kernel.h"
#define SALT 48
#include "CUDA10Kernel.h"
#define SALT 49
#include "CUDA10Kernel.h"
#define SALT 50
#include "CUDA10Kernel.h"
#define SALT 51
#include "CUDA10Kernel.h"
#define SALT 52
#include "CUDA10Kernel.h"
#define SALT 53
#include "CUDA10Kernel.h"
#define SALT 54
#include "CUDA10Kernel.h"
#define SALT 55
#include "CUDA10Kernel.h"
#define SALT 56
#include "CUDA10Kernel.h"
#define SALT 57
#include "CUDA10Kernel.h"
#define SALT 58
#include "CUDA10Kernel.h"
#define SALT 59
#include "CUDA10Kernel.h"
#define SALT 60
#include "CUDA10Kernel.h"
#define SALT 61
#include "CUDA10Kernel.h"
#define SALT 62
#include "CUDA10Kernel.h"
#define SALT 63
#include "CUDA10Kernel.h"
#define SALT 64
#include "CUDA10Kernel.h"
#define SALT 65
#include "CUDA10Kernel.h"
#define SALT 66
#include "CUDA10Kernel.h"
#define SALT 67
#include "CUDA10Kernel.h"
#define SALT 68
#include "CUDA10Kernel.h"
#define SALT 69
#include "CUDA10Kernel.h"
#define SALT 70
#include "CUDA10Kernel.h"
#define SALT 71
#include "CUDA10Kernel.h"
#define SALT 72
#include "CUDA10Kernel.h"
#define SALT 73
#include "CUDA10Kernel.h"
#define SALT 74
#include "CUDA10Kernel.h"
#define SALT 75
#include "CUDA10Kernel.h"
#define SALT 76
#include "CUDA10Kernel.h"
#define SALT 77
#include "CUDA10Kernel.h"
#define SALT 78
#include "CUDA10Kernel.h"
#define SALT 79
#include "CUDA10Kernel.h"
#define SALT 80
#include "CUDA10Kernel.h"
#define SALT 81
#include "CUDA10Kernel.h"
#define SALT 82
#include "CUDA10Kernel.h"
#define SALT 83
#include "CUDA10Kernel.h"
#define SALT 84
#include "CUDA10Kernel.h"
#define SALT 85
#include "CUDA10Kernel.h"
#define SALT 86
#include "CUDA10Kernel.h"
#define SALT 87
#include "CUDA10Kernel.h"
#define SALT 88
#include "CUDA10Kernel.h"
#define SALT 89
#include "CUDA10Kernel.h"
#define SALT 90
#include "CUDA10Kernel.h"
#define SALT 91
#include "CUDA10Kernel.h"
#define SALT 92
#include "CUDA10Kernel.h"
#define SALT 93
#include "CUDA10Kernel.h"
#define SALT 94
#include "CUDA10Kernel.h"
#define SALT 95
#include "CUDA10Kernel.h"
#define SALT 96
#include "CUDA10Kernel.h"
#define SALT 97
#include "CUDA10Kernel.h"
#define SALT 98
#include "CUDA10Kernel.h"
#define SALT 99
#include "CUDA10Kernel.h"
#define SALT 100
#include "CUDA10Kernel.h"
#define SALT 101
#include "CUDA10Kernel.h"
#define SALT 102
#include "CUDA10Kernel.h"
#define SALT 103
#include "CUDA10Kernel.h"
#define SALT 104
#include "CUDA10Kernel.h"
#define SALT 105
#include "CUDA10Kernel.h"
#define SALT 106
#include "CUDA10Kernel.h"
#define SALT 107
#include "CUDA10Kernel.h"
#define SALT 108
#include "CUDA10Kernel.h"
#define SALT 109
#include "CUDA10Kernel.h"
#define SALT 110
#include "CUDA10Kernel.h"
#define SALT 111
#include "CUDA10Kernel.h"
#define SALT 112
#include "CUDA10Kernel.h"
#define SALT 113
#include "CUDA10Kernel.h"
#define SALT 114
#include "CUDA10Kernel.h"
#define SALT 115
#include "CUDA10Kernel.h"
#define SALT 116
#include "CUDA10Kernel.h"
#define SALT 117
#include "CUDA10Kernel.h"
#define SALT 118
#include "CUDA10Kernel.h"
#define SALT 119
#include "CUDA10Kernel.h"
#define SALT 120
#include "CUDA10Kernel.h"
#define SALT 121
#include "CUDA10Kernel.h"
#define SALT 122
#include "CUDA10Kernel.h"
#define SALT 123
#include "CUDA10Kernel.h"
#define SALT 124
#include "CUDA10Kernel.h"
#define SALT 125
#include "CUDA10Kernel.h"
#define SALT 126
#include "CUDA10Kernel.h"
#define SALT 127
#include "CUDA10Kernel.h"
#define SALT 128
#include "CUDA10Kernel.h"
#define SALT 129
#include "CUDA10Kernel.h"
#define SALT 130
#include "CUDA10Kernel.h"
#define SALT 131
#include "CUDA10Kernel.h"
#define SALT 132
#include "CUDA10Kernel.h"
#define SALT 133
#include "CUDA10Kernel.h"
#define SALT 134
#include "CUDA10Kernel.h"
#define SALT 135
#include "CUDA10Kernel.h"
#define SALT 136
#include "CUDA10Kernel.h"
#define SALT 137
#include "CUDA10Kernel.h"
#define SALT 138
#include "CUDA10Kernel.h"
#define SALT 139
#include "CUDA10Kernel.h"
#define SALT 140
#include "CUDA10Kernel.h"
#define SALT 141
#include "CUDA10Kernel.h"
#define SALT 142
#include "CUDA10Kernel.h"
#define SALT 143
#include "CUDA10Kernel.h"
#define SALT 144
#include "CUDA10Kernel.h"
#define SALT 145
#include "CUDA10Kernel.h"
#define SALT 146
#include "CUDA10Kernel.h"
#define SALT 147
#include "CUDA10Kernel.h"
#define SALT 148
#include "CUDA10Kernel.h"
#define SALT 149
#include "CUDA10Kernel.h"
#define SALT 150
#include "CUDA10Kernel.h"
#define SALT 151
#include "CUDA10Kernel.h"
#define SALT 152
#include "CUDA10Kernel.h"
#define SALT 153
#include "CUDA10Kernel.h"
#define SALT 154
#include "CUDA10Kernel.h"
#define SALT 155
#include "CUDA10Kernel.h"
#define SALT 156
#include "CUDA10Kernel.h"
#define SALT 157
#include "CUDA10Kernel.h"
#define SALT 158
#include "CUDA10Kernel.h"
#define SALT 159
#include "CUDA10Kernel.h"
#define SALT 160
#include "CUDA10Kernel.h"
#define SALT 161
#include "CUDA10Kernel.h"
#define SALT 162
#include "CUDA10Kernel.h"
#define SALT 163
#include "CUDA10Kernel.h"
#define SALT 164
#include "CUDA10Kernel.h"
#define SALT 165
#include "CUDA10Kernel.h"
#define SALT 166
#include "CUDA10Kernel.h"
#define SALT 167
#include "CUDA10Kernel.h"
#define SALT 168
#include "CUDA10Kernel.h"
#define SALT 169
#include "CUDA10Kernel.h"
#define SALT 170
#include "CUDA10Kernel.h"
#define SALT 171
#include "CUDA10Kernel.h"
#define SALT 172
#include "CUDA10Kernel.h"
#define SALT 173
#include "CUDA10Kernel.h"
#define SALT 174
#include "CUDA10Kernel.h"
#define SALT 175
#include "CUDA10Kernel.h"
#define SALT 176
#include "CUDA10Kernel.h"
#define SALT 177
#include "CUDA10Kernel.h"
#define SALT 178
#include "CUDA10Kernel.h"
#define SALT 179
#include "CUDA10Kernel.h"
#define SALT 180
#include "CUDA10Kernel.h"
#define SALT 181
#include "CUDA10Kernel.h"
#define SALT 182
#include "CUDA10Kernel.h"
#define SALT 183
#include "CUDA10Kernel.h"
#define SALT 184
#include "CUDA10Kernel.h"
#define SALT 185
#include "CUDA10Kernel.h"
#define SALT 186
#include "CUDA10Kernel.h"
#define SALT 187
#include "CUDA10Kernel.h"
#define SALT 188
#include "CUDA10Kernel.h"
#define SALT 189
#include "CUDA10Kernel.h"
#define SALT 190
#include "CUDA10Kernel.h"
#define SALT 191
#include "CUDA10Kernel.h"
#define SALT 192
#include "CUDA10Kernel.h"
#define SALT 193
#include "CUDA10Kernel.h"
#define SALT 194
#include "CUDA10Kernel.h"
#define SALT 195
#include "CUDA10Kernel.h"
#define SALT 196
#include "CUDA10Kernel.h"
#define SALT 197
#include "CUDA10Kernel.h"
#define SALT 198
#include "CUDA10Kernel.h"
#define SALT 199
#include "CUDA10Kernel.h"
#define SALT 200
#include "CUDA10Kernel.h"
#define SALT 201
#include "CUDA10Kernel.h"
#define SALT 202
#include "CUDA10Kernel.h"
#define SALT 203
#include "CUDA10Kernel.h"
#define SALT 204
#include "CUDA10Kernel.h"
#define SALT 205
#include "CUDA10Kernel.h"
#define SALT 206
#include "CUDA10Kernel.h"
#define SALT 207
#include "CUDA10Kernel.h"
#define SALT 208
#include "CUDA10Kernel.h"
#define SALT 209
#include "CUDA10Kernel.h"
#define SALT 210
#include "CUDA10Kernel.h"
#define SALT 211
#include "CUDA10Kernel.h"
#define SALT 212
#include "CUDA10Kernel.h"
#define SALT 213
#include "CUDA10Kernel.h"
#define SALT 214
#include "CUDA10Kernel.h"
#define SALT 215
#include "CUDA10Kernel.h"
#define SALT 216
#include "CUDA10Kernel.h"
#define SALT 217
#include "CUDA10Kernel.h"
#define SALT 218
#include "CUDA10Kernel.h"
#define SALT 219
#include "CUDA10Kernel.h"
#define SALT 220
#include "CUDA10Kernel.h"
#define SALT 221
#include "CUDA10Kernel.h"
#define SALT 222
#include "CUDA10Kernel.h"
#define SALT 223
#include "CUDA10Kernel.h"
#define SALT 224
#include "CUDA10Kernel.h"
#define SALT 225
#include "CUDA10Kernel.h"
#define SALT 226
#include "CUDA10Kernel.h"
#define SALT 227
#include "CUDA10Kernel.h"
#define SALT 228
#include "CUDA10Kernel.h"
#define SALT 229
#include "CUDA10Kernel.h"
#define SALT 230
#include "CUDA10Kernel.h"
#define SALT 231
#include "CUDA10Kernel.h"
#define SALT 232
#include "CUDA10Kernel.h"
#define SALT 233
#include "CUDA10Kernel.h"
#define SALT 234
#include "CUDA10Kernel.h"
#define SALT 235
#include "CUDA10Kernel.h"
#define SALT 236
#include "CUDA10Kernel.h"
#define SALT 237
#include "CUDA10Kernel.h"
#define SALT 238
#include "CUDA10Kernel.h"
#define SALT 239
#include "CUDA10Kernel.h"
#define SALT 240
#include "CUDA10Kernel.h"
#define SALT 241
#include "CUDA10Kernel.h"
#define SALT 242
#include "CUDA10Kernel.h"
#define SALT 243
#include "CUDA10Kernel.h"
#define SALT 244
#include "CUDA10Kernel.h"
#define SALT 245
#include "CUDA10Kernel.h"
#define SALT 246
#include "CUDA10Kernel.h"
#define SALT 247
#include "CUDA10Kernel.h"
#define SALT 248
#include "CUDA10Kernel.h"
#define SALT 249
#include "CUDA10Kernel.h"
#define SALT 250
#include "CUDA10Kernel.h"
#define SALT 251
#include "CUDA10Kernel.h"
#define SALT 252
#include "CUDA10Kernel.h"
#define SALT 253
#include "CUDA10Kernel.h"
#define SALT 254
#include "CUDA10Kernel.h"
#define SALT 255
#include "CUDA10Kernel.h"
#define SALT 256
#include "CUDA10Kernel.h"
#define SALT 257
#include "CUDA10Kernel.h"
#define SALT 258
#include "CUDA10Kernel.h"
#define SALT 259
#include "CUDA10Kernel.h"
#define SALT 260
#include "CUDA10Kernel.h"
#define SALT 261
#include "CUDA10Kernel.h"
#define SALT 262
#include "CUDA10Kernel.h"
#define SALT 263
#include "CUDA10Kernel.h"
#define SALT 264
#include "CUDA10Kernel.h"
#define SALT 265
#include "CUDA10Kernel.h"
#define SALT 266
#include "CUDA10Kernel.h"
#define SALT 267
#include "CUDA10Kernel.h"
#define SALT 268
#include "CUDA10Kernel.h"
#define SALT 269
#include "CUDA10Kernel.h"
#define SALT 270
#include "CUDA10Kernel.h"
#define SALT 271
#include "CUDA10Kernel.h"
#define SALT 272
#include "CUDA10Kernel.h"
#define SALT 273
#include "CUDA10Kernel.h"
#define SALT 274
#include "CUDA10Kernel.h"
#define SALT 275
#include "CUDA10Kernel.h"
#define SALT 276
#include "CUDA10Kernel.h"
#define SALT 277
#include "CUDA10Kernel.h"
#define SALT 278
#include "CUDA10Kernel.h"
#define SALT 279
#include "CUDA10Kernel.h"
#define SALT 280
#include "CUDA10Kernel.h"
#define SALT 281
#include "CUDA10Kernel.h"
#define SALT 282
#include "CUDA10Kernel.h"
#define SALT 283
#include "CUDA10Kernel.h"
#define SALT 284
#include "CUDA10Kernel.h"
#define SALT 285
#include "CUDA10Kernel.h"
#define SALT 286
#include "CUDA10Kernel.h"
#define SALT 287
#include "CUDA10Kernel.h"
#define SALT 288
#include "CUDA10Kernel.h"
#define SALT 289
#include "CUDA10Kernel.h"
#define SALT 290
#include "CUDA10Kernel.h"
#define SALT 291
#include "CUDA10Kernel.h"
#define SALT 292
#include "CUDA10Kernel.h"
#define SALT 293
#include "CUDA10Kernel.h"
#define SALT 294
#include "CUDA10Kernel.h"
#define SALT 295
#include "CUDA10Kernel.h"
#define SALT 296
#include "CUDA10Kernel.h"
#define SALT 297
#include "CUDA10Kernel.h"
#define SALT 298
#include "CUDA10Kernel.h"
#define SALT 299
#include "CUDA10Kernel.h"
#define SALT 300
#include "CUDA10Kernel.h"
#define SALT 301
#include "CUDA10Kernel.h"
#define SALT 302
#include "CUDA10Kernel.h"
#define SALT 303
#include "CUDA10Kernel.h"
#define SALT 304
#include "CUDA10Kernel.h"
#define SALT 305
#include "CUDA10Kernel.h"
#define SALT 306
#include "CUDA10Kernel.h"
#define SALT 307
#include "CUDA10Kernel.h"
#define SALT 308
#include "CUDA10Kernel.h"
#define SALT 309
#include "CUDA10Kernel.h"
#define SALT 310
#include "CUDA10Kernel.h"
#define SALT 311
#include "CUDA10Kernel.h"
#define SALT 312
#include "CUDA10Kernel.h"
#define SALT 313
#include "CUDA10Kernel.h"
#define SALT 314
#include "CUDA10Kernel.h"
#define SALT 315
#include "CUDA10Kernel.h"
#define SALT 316
#include "CUDA10Kernel.h"
#define SALT 317
#include "CUDA10Kernel.h"
#define SALT 318
#include "CUDA10Kernel.h"
#define SALT 319
#include "CUDA10Kernel.h"
#define SALT 320
#include "CUDA10Kernel.h"
#define SALT 321
#include "CUDA10Kernel.h"
#define SALT 322
#include "CUDA10Kernel.h"
#define SALT 323
#include "CUDA10Kernel.h"
#define SALT 324
#include "CUDA10Kernel.h"
#define SALT 325
#include "CUDA10Kernel.h"
#define SALT 326
#include "CUDA10Kernel.h"
#define SALT 327
#include "CUDA10Kernel.h"
#define SALT 328
#include "CUDA10Kernel.h"
#define SALT 329
#include "CUDA10Kernel.h"
#define SALT 330
#include "CUDA10Kernel.h"
#define SALT 331
#include "CUDA10Kernel.h"
#define SALT 332
#include "CUDA10Kernel.h"
#define SALT 333
#include "CUDA10Kernel.h"
#define SALT 334
#include "CUDA10Kernel.h"
#define SALT 335
#include "CUDA10Kernel.h"
#define SALT 336
#include "CUDA10Kernel.h"
#define SALT 337
#include "CUDA10Kernel.h"
#define SALT 338
#include "CUDA10Kernel.h"
#define SALT 339
#include "CUDA10Kernel.h"
#define SALT 340
#include "CUDA10Kernel.h"
#define SALT 341
#include "CUDA10Kernel.h"
#define SALT 342
#include "CUDA10Kernel.h"
#define SALT 343
#include "CUDA10Kernel.h"
#define SALT 344
#include "CUDA10Kernel.h"
#define SALT 345
#include "CUDA10Kernel.h"
#define SALT 346
#include "CUDA10Kernel.h"
#define SALT 347
#include "CUDA10Kernel.h"
#define SALT 348
#include "CUDA10Kernel.h"
#define SALT 349
#include "CUDA10Kernel.h"
#define SALT 350
#include "CUDA10Kernel.h"
#define SALT 351
#include "CUDA10Kernel.h"
#define SALT 352
#include "CUDA10Kernel.h"
#define SALT 353
#include "CUDA10Kernel.h"
#define SALT 354
#include "CUDA10Kernel.h"
#define SALT 355
#include "CUDA10Kernel.h"
#define SALT 356
#include "CUDA10Kernel.h"
#define SALT 357
#include "CUDA10Kernel.h"
#define SALT 358
#include "CUDA10Kernel.h"
#define SALT 359
#include "CUDA10Kernel.h"
#define SALT 360
#include "CUDA10Kernel.h"
#define SALT 361
#include "CUDA10Kernel.h"
#define SALT 362
#include "CUDA10Kernel.h"
#define SALT 363
#include "CUDA10Kernel.h"
#define SALT 364
#include "CUDA10Kernel.h"
#define SALT 365
#include "CUDA10Kernel.h"
#define SALT 366
#include "CUDA10Kernel.h"
#define SALT 367
#include "CUDA10Kernel.h"
#define SALT 368
#include "CUDA10Kernel.h"
#define SALT 369
#include "CUDA10Kernel.h"
#define SALT 370
#include "CUDA10Kernel.h"
#define SALT 371
#include "CUDA10Kernel.h"
#define SALT 372
#include "CUDA10Kernel.h"
#define SALT 373
#include "CUDA10Kernel.h"
#define SALT 374
#include "CUDA10Kernel.h"
#define SALT 375
#include "CUDA10Kernel.h"
#define SALT 376
#include "CUDA10Kernel.h"
#define SALT 377
#include "CUDA10Kernel.h"
#define SALT 378
#include "CUDA10Kernel.h"
#define SALT 379
#include "CUDA10Kernel.h"
#define SALT 380
#include "CUDA10Kernel.h"
#define SALT 381
#include "CUDA10Kernel.h"
#define SALT 382
#include "CUDA10Kernel.h"
#define SALT 383
#include "CUDA10Kernel.h"
#define SALT 384
#include "CUDA10Kernel.h"
#define SALT 385
#include "CUDA10Kernel.h"
#define SALT 386
#include "CUDA10Kernel.h"
#define SALT 387
#include "CUDA10Kernel.h"
#define SALT 388
#include "CUDA10Kernel.h"
#define SALT 389
#include "CUDA10Kernel.h"
#define SALT 390
#include "CUDA10Kernel.h"
#define SALT 391
#include "CUDA10Kernel.h"
#define SALT 392
#include "CUDA10Kernel.h"
#define SALT 393
#include "CUDA10Kernel.h"
#define SALT 394
#include "CUDA10Kernel.h"
#define SALT 395
#include "CUDA10Kernel.h"
#define SALT 396
#include "CUDA10Kernel.h"
#define SALT 397
#include "CUDA10Kernel.h"
#define SALT 398
#include "CUDA10Kernel.h"
#define SALT 399
#include "CUDA10Kernel.h"
#define SALT 400
#include "CUDA10Kernel.h"
#define SALT 401
#include "CUDA10Kernel.h"
#define SALT 402
#include "CUDA10Kernel.h"
#define SALT 403
#include "CUDA10Kernel.h"
#define SALT 404
#include "CUDA10Kernel.h"
#define SALT 405
#include "CUDA10Kernel.h"
#define SALT 406
#include "CUDA10Kernel.h"
#define SALT 407
#include "CUDA10Kernel.h"
#define SALT 408
#include "CUDA10Kernel.h"
#define SALT 409
#include "CUDA10Kernel.h"
#define SALT 410
#include "CUDA10Kernel.h"
#define SALT 411
#include "CUDA10Kernel.h"
#define SALT 412
#include "CUDA10Kernel.h"
#define SALT 413
#include "CUDA10Kernel.h"
#define SALT 414
#include "CUDA10Kernel.h"
#define SALT 415
#include "CUDA10Kernel.h"
#define SALT 416
#include "CUDA10Kernel.h"
#define SALT 417
#include "CUDA10Kernel.h"
#define SALT 418
#include "CUDA10Kernel.h"
#define SALT 419
#include "CUDA10Kernel.h"
#define SALT 420
#include "CUDA10Kernel.h"
#define SALT 421
#include "CUDA10Kernel.h"
#define SALT 422
#include "CUDA10Kernel.h"
#define SALT 423
#include "CUDA10Kernel.h"
#define SALT 424
#include "CUDA10Kernel.h"
#define SALT 425
#include "CUDA10Kernel.h"
#define SALT 426
#include "CUDA10Kernel.h"
#define SALT 427
#include "CUDA10Kernel.h"
#define SALT 428
#include "CUDA10Kernel.h"
#define SALT 429
#include "CUDA10Kernel.h"
#define SALT 430
#include "CUDA10Kernel.h"
#define SALT 431
#include "CUDA10Kernel.h"
#define SALT 432
#include "CUDA10Kernel.h"
#define SALT 433
#include "CUDA10Kernel.h"
#define SALT 434
#include "CUDA10Kernel.h"
#define SALT 435
#include "CUDA10Kernel.h"
#define SALT 436
#include "CUDA10Kernel.h"
#define SALT 437
#include "CUDA10Kernel.h"
#define SALT 438
#include "CUDA10Kernel.h"
#define SALT 439
#include "CUDA10Kernel.h"
#define SALT 440
#include "CUDA10Kernel.h"
#define SALT 441
#include "CUDA10Kernel.h"
#define SALT 442
#include "CUDA10Kernel.h"
#define SALT 443
#include "CUDA10Kernel.h"
#define SALT 444
#include "CUDA10Kernel.h"
#define SALT 445
#include "CUDA10Kernel.h"
#define SALT 446
#include "CUDA10Kernel.h"
#define SALT 447
#include "CUDA10Kernel.h"
#define SALT 448
#include "CUDA10Kernel.h"
#define SALT 449
#include "CUDA10Kernel.h"
#define SALT 450
#include "CUDA10Kernel.h"
#define SALT 451
#include "CUDA10Kernel.h"
#define SALT 452
#include "CUDA10Kernel.h"
#define SALT 453
#include "CUDA10Kernel.h"
#define SALT 454
#include "CUDA10Kernel.h"
#define SALT 455
#include "CUDA10Kernel.h"
#define SALT 456
#include "CUDA10Kernel.h"
#define SALT 457
#include "CUDA10Kernel.h"
#define SALT 458
#include "CUDA10Kernel.h"
#define SALT 459
#include "CUDA10Kernel.h"
#define SALT 460
#include "CUDA10Kernel.h"
#define SALT 461
#include "CUDA10Kernel.h"
#define SALT 462
#include "CUDA10Kernel.h"
#define SALT 463
#include "CUDA10Kernel.h"
#define SALT 464
#include "CUDA10Kernel.h"
#define SALT 465
#include "CUDA10Kernel.h"
#define SALT 466
#include "CUDA10Kernel.h"
#define SALT 467
#include "CUDA10Kernel.h"
#define SALT 468
#include "CUDA10Kernel.h"
#define SALT 469
#include "CUDA10Kernel.h"
#define SALT 470
#include "CUDA10Kernel.h"
#define SALT 471
#include "CUDA10Kernel.h"
#define SALT 472
#include "CUDA10Kernel.h"
#define SALT 473
#include "CUDA10Kernel.h"
#define SALT 474
#include "CUDA10Kernel.h"
#define SALT 475
#include "CUDA10Kernel.h"
#define SALT 476
#include "CUDA10Kernel.h"
#define SALT 477
#include "CUDA10Kernel.h"
#define SALT 478
#include "CUDA10Kernel.h"
#define SALT 479
#include "CUDA10Kernel.h"
#define SALT 480
#include "CUDA10Kernel.h"
#define SALT 481
#include "CUDA10Kernel.h"
#define SALT 482
#include "CUDA10Kernel.h"
#define SALT 483
#include "CUDA10Kernel.h"
#define SALT 484
#include "CUDA10Kernel.h"
#define SALT 485
#include "CUDA10Kernel.h"
#define SALT 486
#include "CUDA10Kernel.h"
#define SALT 487
#include "CUDA10Kernel.h"
#define SALT 488
#include "CUDA10Kernel.h"
#define SALT 489
#include "CUDA10Kernel.h"
#define SALT 490
#include "CUDA10Kernel.h"
#define SALT 491
#include "CUDA10Kernel.h"
#define SALT 492
#include "CUDA10Kernel.h"
#define SALT 493
#include "CUDA10Kernel.h"
#define SALT 494
#include "CUDA10Kernel.h"
#define SALT 495
#include "CUDA10Kernel.h"
#define SALT 496
#include "CUDA10Kernel.h"
#define SALT 497
#include "CUDA10Kernel.h"
#define SALT 498
#include "CUDA10Kernel.h"
#define SALT 499
#include "CUDA10Kernel.h"
#define SALT 500
#include "CUDA10Kernel.h"
#define SALT 501
#include "CUDA10Kernel.h"
#define SALT 502
#include "CUDA10Kernel.h"
#define SALT 503
#include "CUDA10Kernel.h"
#define SALT 504
#include "CUDA10Kernel.h"
#define SALT 505
#include "CUDA10Kernel.h"
#define SALT 506
#include "CUDA10Kernel.h"
#define SALT 507
#include "CUDA10Kernel.h"
#define SALT 508
#include "CUDA10Kernel.h"
#define SALT 509
#include "CUDA10Kernel.h"
#define SALT 510
#include "CUDA10Kernel.h"
#define SALT 511
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher0()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel0(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 0: CUDA_DES_PerformSearch0<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1: CUDA_DES_PerformSearch1<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2: CUDA_DES_PerformSearch2<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3: CUDA_DES_PerformSearch3<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4: CUDA_DES_PerformSearch4<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 5: CUDA_DES_PerformSearch5<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 6: CUDA_DES_PerformSearch6<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 7: CUDA_DES_PerformSearch7<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 8: CUDA_DES_PerformSearch8<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 9: CUDA_DES_PerformSearch9<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 10: CUDA_DES_PerformSearch10<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 11: CUDA_DES_PerformSearch11<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 12: CUDA_DES_PerformSearch12<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 13: CUDA_DES_PerformSearch13<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 14: CUDA_DES_PerformSearch14<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 15: CUDA_DES_PerformSearch15<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 16: CUDA_DES_PerformSearch16<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 17: CUDA_DES_PerformSearch17<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 18: CUDA_DES_PerformSearch18<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 19: CUDA_DES_PerformSearch19<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 20: CUDA_DES_PerformSearch20<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 21: CUDA_DES_PerformSearch21<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 22: CUDA_DES_PerformSearch22<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 23: CUDA_DES_PerformSearch23<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 24: CUDA_DES_PerformSearch24<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 25: CUDA_DES_PerformSearch25<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 26: CUDA_DES_PerformSearch26<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 27: CUDA_DES_PerformSearch27<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 28: CUDA_DES_PerformSearch28<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 29: CUDA_DES_PerformSearch29<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 30: CUDA_DES_PerformSearch30<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 31: CUDA_DES_PerformSearch31<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 32: CUDA_DES_PerformSearch32<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 33: CUDA_DES_PerformSearch33<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 34: CUDA_DES_PerformSearch34<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 35: CUDA_DES_PerformSearch35<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 36: CUDA_DES_PerformSearch36<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 37: CUDA_DES_PerformSearch37<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 38: CUDA_DES_PerformSearch38<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 39: CUDA_DES_PerformSearch39<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 40: CUDA_DES_PerformSearch40<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 41: CUDA_DES_PerformSearch41<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 42: CUDA_DES_PerformSearch42<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 43: CUDA_DES_PerformSearch43<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 44: CUDA_DES_PerformSearch44<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 45: CUDA_DES_PerformSearch45<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 46: CUDA_DES_PerformSearch46<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 47: CUDA_DES_PerformSearch47<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 48: CUDA_DES_PerformSearch48<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 49: CUDA_DES_PerformSearch49<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 50: CUDA_DES_PerformSearch50<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 51: CUDA_DES_PerformSearch51<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 52: CUDA_DES_PerformSearch52<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 53: CUDA_DES_PerformSearch53<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 54: CUDA_DES_PerformSearch54<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 55: CUDA_DES_PerformSearch55<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 56: CUDA_DES_PerformSearch56<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 57: CUDA_DES_PerformSearch57<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 58: CUDA_DES_PerformSearch58<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 59: CUDA_DES_PerformSearch59<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 60: CUDA_DES_PerformSearch60<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 61: CUDA_DES_PerformSearch61<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 62: CUDA_DES_PerformSearch62<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 63: CUDA_DES_PerformSearch63<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 64: CUDA_DES_PerformSearch64<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 65: CUDA_DES_PerformSearch65<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 66: CUDA_DES_PerformSearch66<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 67: CUDA_DES_PerformSearch67<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 68: CUDA_DES_PerformSearch68<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 69: CUDA_DES_PerformSearch69<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 70: CUDA_DES_PerformSearch70<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 71: CUDA_DES_PerformSearch71<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 72: CUDA_DES_PerformSearch72<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 73: CUDA_DES_PerformSearch73<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 74: CUDA_DES_PerformSearch74<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 75: CUDA_DES_PerformSearch75<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 76: CUDA_DES_PerformSearch76<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 77: CUDA_DES_PerformSearch77<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 78: CUDA_DES_PerformSearch78<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 79: CUDA_DES_PerformSearch79<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 80: CUDA_DES_PerformSearch80<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 81: CUDA_DES_PerformSearch81<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 82: CUDA_DES_PerformSearch82<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 83: CUDA_DES_PerformSearch83<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 84: CUDA_DES_PerformSearch84<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 85: CUDA_DES_PerformSearch85<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 86: CUDA_DES_PerformSearch86<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 87: CUDA_DES_PerformSearch87<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 88: CUDA_DES_PerformSearch88<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 89: CUDA_DES_PerformSearch89<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 90: CUDA_DES_PerformSearch90<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 91: CUDA_DES_PerformSearch91<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 92: CUDA_DES_PerformSearch92<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 93: CUDA_DES_PerformSearch93<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 94: CUDA_DES_PerformSearch94<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 95: CUDA_DES_PerformSearch95<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 96: CUDA_DES_PerformSearch96<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 97: CUDA_DES_PerformSearch97<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 98: CUDA_DES_PerformSearch98<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 99: CUDA_DES_PerformSearch99<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 100: CUDA_DES_PerformSearch100<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 101: CUDA_DES_PerformSearch101<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 102: CUDA_DES_PerformSearch102<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 103: CUDA_DES_PerformSearch103<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 104: CUDA_DES_PerformSearch104<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 105: CUDA_DES_PerformSearch105<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 106: CUDA_DES_PerformSearch106<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 107: CUDA_DES_PerformSearch107<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 108: CUDA_DES_PerformSearch108<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 109: CUDA_DES_PerformSearch109<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 110: CUDA_DES_PerformSearch110<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 111: CUDA_DES_PerformSearch111<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 112: CUDA_DES_PerformSearch112<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 113: CUDA_DES_PerformSearch113<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 114: CUDA_DES_PerformSearch114<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 115: CUDA_DES_PerformSearch115<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 116: CUDA_DES_PerformSearch116<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 117: CUDA_DES_PerformSearch117<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 118: CUDA_DES_PerformSearch118<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 119: CUDA_DES_PerformSearch119<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 120: CUDA_DES_PerformSearch120<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 121: CUDA_DES_PerformSearch121<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 122: CUDA_DES_PerformSearch122<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 123: CUDA_DES_PerformSearch123<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 124: CUDA_DES_PerformSearch124<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 125: CUDA_DES_PerformSearch125<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 126: CUDA_DES_PerformSearch126<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 127: CUDA_DES_PerformSearch127<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 128: CUDA_DES_PerformSearch128<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 129: CUDA_DES_PerformSearch129<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 130: CUDA_DES_PerformSearch130<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 131: CUDA_DES_PerformSearch131<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 132: CUDA_DES_PerformSearch132<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 133: CUDA_DES_PerformSearch133<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 134: CUDA_DES_PerformSearch134<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 135: CUDA_DES_PerformSearch135<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 136: CUDA_DES_PerformSearch136<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 137: CUDA_DES_PerformSearch137<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 138: CUDA_DES_PerformSearch138<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 139: CUDA_DES_PerformSearch139<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 140: CUDA_DES_PerformSearch140<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 141: CUDA_DES_PerformSearch141<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 142: CUDA_DES_PerformSearch142<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 143: CUDA_DES_PerformSearch143<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 144: CUDA_DES_PerformSearch144<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 145: CUDA_DES_PerformSearch145<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 146: CUDA_DES_PerformSearch146<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 147: CUDA_DES_PerformSearch147<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 148: CUDA_DES_PerformSearch148<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 149: CUDA_DES_PerformSearch149<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 150: CUDA_DES_PerformSearch150<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 151: CUDA_DES_PerformSearch151<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 152: CUDA_DES_PerformSearch152<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 153: CUDA_DES_PerformSearch153<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 154: CUDA_DES_PerformSearch154<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 155: CUDA_DES_PerformSearch155<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 156: CUDA_DES_PerformSearch156<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 157: CUDA_DES_PerformSearch157<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 158: CUDA_DES_PerformSearch158<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 159: CUDA_DES_PerformSearch159<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 160: CUDA_DES_PerformSearch160<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 161: CUDA_DES_PerformSearch161<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 162: CUDA_DES_PerformSearch162<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 163: CUDA_DES_PerformSearch163<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 164: CUDA_DES_PerformSearch164<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 165: CUDA_DES_PerformSearch165<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 166: CUDA_DES_PerformSearch166<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 167: CUDA_DES_PerformSearch167<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 168: CUDA_DES_PerformSearch168<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 169: CUDA_DES_PerformSearch169<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 170: CUDA_DES_PerformSearch170<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 171: CUDA_DES_PerformSearch171<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 172: CUDA_DES_PerformSearch172<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 173: CUDA_DES_PerformSearch173<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 174: CUDA_DES_PerformSearch174<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 175: CUDA_DES_PerformSearch175<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 176: CUDA_DES_PerformSearch176<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 177: CUDA_DES_PerformSearch177<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 178: CUDA_DES_PerformSearch178<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 179: CUDA_DES_PerformSearch179<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 180: CUDA_DES_PerformSearch180<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 181: CUDA_DES_PerformSearch181<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 182: CUDA_DES_PerformSearch182<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 183: CUDA_DES_PerformSearch183<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 184: CUDA_DES_PerformSearch184<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 185: CUDA_DES_PerformSearch185<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 186: CUDA_DES_PerformSearch186<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 187: CUDA_DES_PerformSearch187<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 188: CUDA_DES_PerformSearch188<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 189: CUDA_DES_PerformSearch189<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 190: CUDA_DES_PerformSearch190<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 191: CUDA_DES_PerformSearch191<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 192: CUDA_DES_PerformSearch192<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 193: CUDA_DES_PerformSearch193<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 194: CUDA_DES_PerformSearch194<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 195: CUDA_DES_PerformSearch195<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 196: CUDA_DES_PerformSearch196<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 197: CUDA_DES_PerformSearch197<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 198: CUDA_DES_PerformSearch198<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 199: CUDA_DES_PerformSearch199<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 200: CUDA_DES_PerformSearch200<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 201: CUDA_DES_PerformSearch201<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 202: CUDA_DES_PerformSearch202<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 203: CUDA_DES_PerformSearch203<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 204: CUDA_DES_PerformSearch204<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 205: CUDA_DES_PerformSearch205<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 206: CUDA_DES_PerformSearch206<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 207: CUDA_DES_PerformSearch207<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 208: CUDA_DES_PerformSearch208<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 209: CUDA_DES_PerformSearch209<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 210: CUDA_DES_PerformSearch210<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 211: CUDA_DES_PerformSearch211<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 212: CUDA_DES_PerformSearch212<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 213: CUDA_DES_PerformSearch213<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 214: CUDA_DES_PerformSearch214<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 215: CUDA_DES_PerformSearch215<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 216: CUDA_DES_PerformSearch216<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 217: CUDA_DES_PerformSearch217<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 218: CUDA_DES_PerformSearch218<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 219: CUDA_DES_PerformSearch219<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 220: CUDA_DES_PerformSearch220<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 221: CUDA_DES_PerformSearch221<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 222: CUDA_DES_PerformSearch222<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 223: CUDA_DES_PerformSearch223<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 224: CUDA_DES_PerformSearch224<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 225: CUDA_DES_PerformSearch225<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 226: CUDA_DES_PerformSearch226<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 227: CUDA_DES_PerformSearch227<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 228: CUDA_DES_PerformSearch228<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 229: CUDA_DES_PerformSearch229<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 230: CUDA_DES_PerformSearch230<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 231: CUDA_DES_PerformSearch231<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 232: CUDA_DES_PerformSearch232<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 233: CUDA_DES_PerformSearch233<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 234: CUDA_DES_PerformSearch234<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 235: CUDA_DES_PerformSearch235<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 236: CUDA_DES_PerformSearch236<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 237: CUDA_DES_PerformSearch237<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 238: CUDA_DES_PerformSearch238<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 239: CUDA_DES_PerformSearch239<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 240: CUDA_DES_PerformSearch240<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 241: CUDA_DES_PerformSearch241<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 242: CUDA_DES_PerformSearch242<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 243: CUDA_DES_PerformSearch243<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 244: CUDA_DES_PerformSearch244<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 245: CUDA_DES_PerformSearch245<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 246: CUDA_DES_PerformSearch246<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 247: CUDA_DES_PerformSearch247<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 248: CUDA_DES_PerformSearch248<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 249: CUDA_DES_PerformSearch249<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 250: CUDA_DES_PerformSearch250<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 251: CUDA_DES_PerformSearch251<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 252: CUDA_DES_PerformSearch252<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 253: CUDA_DES_PerformSearch253<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 254: CUDA_DES_PerformSearch254<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 255: CUDA_DES_PerformSearch255<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 256: CUDA_DES_PerformSearch256<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 257: CUDA_DES_PerformSearch257<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 258: CUDA_DES_PerformSearch258<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 259: CUDA_DES_PerformSearch259<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 260: CUDA_DES_PerformSearch260<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 261: CUDA_DES_PerformSearch261<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 262: CUDA_DES_PerformSearch262<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 263: CUDA_DES_PerformSearch263<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 264: CUDA_DES_PerformSearch264<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 265: CUDA_DES_PerformSearch265<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 266: CUDA_DES_PerformSearch266<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 267: CUDA_DES_PerformSearch267<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 268: CUDA_DES_PerformSearch268<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 269: CUDA_DES_PerformSearch269<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 270: CUDA_DES_PerformSearch270<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 271: CUDA_DES_PerformSearch271<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 272: CUDA_DES_PerformSearch272<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 273: CUDA_DES_PerformSearch273<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 274: CUDA_DES_PerformSearch274<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 275: CUDA_DES_PerformSearch275<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 276: CUDA_DES_PerformSearch276<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 277: CUDA_DES_PerformSearch277<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 278: CUDA_DES_PerformSearch278<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 279: CUDA_DES_PerformSearch279<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 280: CUDA_DES_PerformSearch280<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 281: CUDA_DES_PerformSearch281<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 282: CUDA_DES_PerformSearch282<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 283: CUDA_DES_PerformSearch283<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 284: CUDA_DES_PerformSearch284<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 285: CUDA_DES_PerformSearch285<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 286: CUDA_DES_PerformSearch286<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 287: CUDA_DES_PerformSearch287<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 288: CUDA_DES_PerformSearch288<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 289: CUDA_DES_PerformSearch289<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 290: CUDA_DES_PerformSearch290<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 291: CUDA_DES_PerformSearch291<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 292: CUDA_DES_PerformSearch292<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 293: CUDA_DES_PerformSearch293<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 294: CUDA_DES_PerformSearch294<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 295: CUDA_DES_PerformSearch295<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 296: CUDA_DES_PerformSearch296<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 297: CUDA_DES_PerformSearch297<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 298: CUDA_DES_PerformSearch298<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 299: CUDA_DES_PerformSearch299<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 300: CUDA_DES_PerformSearch300<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 301: CUDA_DES_PerformSearch301<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 302: CUDA_DES_PerformSearch302<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 303: CUDA_DES_PerformSearch303<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 304: CUDA_DES_PerformSearch304<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 305: CUDA_DES_PerformSearch305<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 306: CUDA_DES_PerformSearch306<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 307: CUDA_DES_PerformSearch307<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 308: CUDA_DES_PerformSearch308<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 309: CUDA_DES_PerformSearch309<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 310: CUDA_DES_PerformSearch310<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 311: CUDA_DES_PerformSearch311<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 312: CUDA_DES_PerformSearch312<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 313: CUDA_DES_PerformSearch313<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 314: CUDA_DES_PerformSearch314<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 315: CUDA_DES_PerformSearch315<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 316: CUDA_DES_PerformSearch316<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 317: CUDA_DES_PerformSearch317<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 318: CUDA_DES_PerformSearch318<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 319: CUDA_DES_PerformSearch319<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 320: CUDA_DES_PerformSearch320<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 321: CUDA_DES_PerformSearch321<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 322: CUDA_DES_PerformSearch322<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 323: CUDA_DES_PerformSearch323<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 324: CUDA_DES_PerformSearch324<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 325: CUDA_DES_PerformSearch325<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 326: CUDA_DES_PerformSearch326<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 327: CUDA_DES_PerformSearch327<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 328: CUDA_DES_PerformSearch328<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 329: CUDA_DES_PerformSearch329<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 330: CUDA_DES_PerformSearch330<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 331: CUDA_DES_PerformSearch331<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 332: CUDA_DES_PerformSearch332<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 333: CUDA_DES_PerformSearch333<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 334: CUDA_DES_PerformSearch334<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 335: CUDA_DES_PerformSearch335<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 336: CUDA_DES_PerformSearch336<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 337: CUDA_DES_PerformSearch337<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 338: CUDA_DES_PerformSearch338<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 339: CUDA_DES_PerformSearch339<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 340: CUDA_DES_PerformSearch340<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 341: CUDA_DES_PerformSearch341<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 342: CUDA_DES_PerformSearch342<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 343: CUDA_DES_PerformSearch343<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 344: CUDA_DES_PerformSearch344<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 345: CUDA_DES_PerformSearch345<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 346: CUDA_DES_PerformSearch346<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 347: CUDA_DES_PerformSearch347<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 348: CUDA_DES_PerformSearch348<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 349: CUDA_DES_PerformSearch349<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 350: CUDA_DES_PerformSearch350<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 351: CUDA_DES_PerformSearch351<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 352: CUDA_DES_PerformSearch352<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 353: CUDA_DES_PerformSearch353<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 354: CUDA_DES_PerformSearch354<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 355: CUDA_DES_PerformSearch355<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 356: CUDA_DES_PerformSearch356<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 357: CUDA_DES_PerformSearch357<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 358: CUDA_DES_PerformSearch358<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 359: CUDA_DES_PerformSearch359<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 360: CUDA_DES_PerformSearch360<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 361: CUDA_DES_PerformSearch361<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 362: CUDA_DES_PerformSearch362<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 363: CUDA_DES_PerformSearch363<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 364: CUDA_DES_PerformSearch364<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 365: CUDA_DES_PerformSearch365<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 366: CUDA_DES_PerformSearch366<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 367: CUDA_DES_PerformSearch367<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 368: CUDA_DES_PerformSearch368<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 369: CUDA_DES_PerformSearch369<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 370: CUDA_DES_PerformSearch370<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 371: CUDA_DES_PerformSearch371<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 372: CUDA_DES_PerformSearch372<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 373: CUDA_DES_PerformSearch373<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 374: CUDA_DES_PerformSearch374<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 375: CUDA_DES_PerformSearch375<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 376: CUDA_DES_PerformSearch376<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 377: CUDA_DES_PerformSearch377<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 378: CUDA_DES_PerformSearch378<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 379: CUDA_DES_PerformSearch379<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 380: CUDA_DES_PerformSearch380<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 381: CUDA_DES_PerformSearch381<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 382: CUDA_DES_PerformSearch382<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 383: CUDA_DES_PerformSearch383<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 384: CUDA_DES_PerformSearch384<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 385: CUDA_DES_PerformSearch385<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 386: CUDA_DES_PerformSearch386<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 387: CUDA_DES_PerformSearch387<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 388: CUDA_DES_PerformSearch388<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 389: CUDA_DES_PerformSearch389<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 390: CUDA_DES_PerformSearch390<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 391: CUDA_DES_PerformSearch391<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 392: CUDA_DES_PerformSearch392<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 393: CUDA_DES_PerformSearch393<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 394: CUDA_DES_PerformSearch394<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 395: CUDA_DES_PerformSearch395<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 396: CUDA_DES_PerformSearch396<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 397: CUDA_DES_PerformSearch397<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 398: CUDA_DES_PerformSearch398<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 399: CUDA_DES_PerformSearch399<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 400: CUDA_DES_PerformSearch400<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 401: CUDA_DES_PerformSearch401<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 402: CUDA_DES_PerformSearch402<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 403: CUDA_DES_PerformSearch403<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 404: CUDA_DES_PerformSearch404<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 405: CUDA_DES_PerformSearch405<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 406: CUDA_DES_PerformSearch406<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 407: CUDA_DES_PerformSearch407<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 408: CUDA_DES_PerformSearch408<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 409: CUDA_DES_PerformSearch409<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 410: CUDA_DES_PerformSearch410<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 411: CUDA_DES_PerformSearch411<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 412: CUDA_DES_PerformSearch412<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 413: CUDA_DES_PerformSearch413<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 414: CUDA_DES_PerformSearch414<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 415: CUDA_DES_PerformSearch415<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 416: CUDA_DES_PerformSearch416<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 417: CUDA_DES_PerformSearch417<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 418: CUDA_DES_PerformSearch418<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 419: CUDA_DES_PerformSearch419<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 420: CUDA_DES_PerformSearch420<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 421: CUDA_DES_PerformSearch421<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 422: CUDA_DES_PerformSearch422<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 423: CUDA_DES_PerformSearch423<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 424: CUDA_DES_PerformSearch424<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 425: CUDA_DES_PerformSearch425<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 426: CUDA_DES_PerformSearch426<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 427: CUDA_DES_PerformSearch427<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 428: CUDA_DES_PerformSearch428<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 429: CUDA_DES_PerformSearch429<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 430: CUDA_DES_PerformSearch430<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 431: CUDA_DES_PerformSearch431<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 432: CUDA_DES_PerformSearch432<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 433: CUDA_DES_PerformSearch433<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 434: CUDA_DES_PerformSearch434<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 435: CUDA_DES_PerformSearch435<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 436: CUDA_DES_PerformSearch436<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 437: CUDA_DES_PerformSearch437<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 438: CUDA_DES_PerformSearch438<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 439: CUDA_DES_PerformSearch439<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 440: CUDA_DES_PerformSearch440<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 441: CUDA_DES_PerformSearch441<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 442: CUDA_DES_PerformSearch442<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 443: CUDA_DES_PerformSearch443<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 444: CUDA_DES_PerformSearch444<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 445: CUDA_DES_PerformSearch445<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 446: CUDA_DES_PerformSearch446<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 447: CUDA_DES_PerformSearch447<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 448: CUDA_DES_PerformSearch448<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 449: CUDA_DES_PerformSearch449<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 450: CUDA_DES_PerformSearch450<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 451: CUDA_DES_PerformSearch451<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 452: CUDA_DES_PerformSearch452<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 453: CUDA_DES_PerformSearch453<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 454: CUDA_DES_PerformSearch454<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 455: CUDA_DES_PerformSearch455<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 456: CUDA_DES_PerformSearch456<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 457: CUDA_DES_PerformSearch457<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 458: CUDA_DES_PerformSearch458<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 459: CUDA_DES_PerformSearch459<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 460: CUDA_DES_PerformSearch460<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 461: CUDA_DES_PerformSearch461<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 462: CUDA_DES_PerformSearch462<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 463: CUDA_DES_PerformSearch463<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 464: CUDA_DES_PerformSearch464<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 465: CUDA_DES_PerformSearch465<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 466: CUDA_DES_PerformSearch466<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 467: CUDA_DES_PerformSearch467<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 468: CUDA_DES_PerformSearch468<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 469: CUDA_DES_PerformSearch469<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 470: CUDA_DES_PerformSearch470<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 471: CUDA_DES_PerformSearch471<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 472: CUDA_DES_PerformSearch472<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 473: CUDA_DES_PerformSearch473<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 474: CUDA_DES_PerformSearch474<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 475: CUDA_DES_PerformSearch475<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 476: CUDA_DES_PerformSearch476<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 477: CUDA_DES_PerformSearch477<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 478: CUDA_DES_PerformSearch478<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 479: CUDA_DES_PerformSearch479<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 480: CUDA_DES_PerformSearch480<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 481: CUDA_DES_PerformSearch481<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 482: CUDA_DES_PerformSearch482<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 483: CUDA_DES_PerformSearch483<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 484: CUDA_DES_PerformSearch484<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 485: CUDA_DES_PerformSearch485<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 486: CUDA_DES_PerformSearch486<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 487: CUDA_DES_PerformSearch487<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 488: CUDA_DES_PerformSearch488<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 489: CUDA_DES_PerformSearch489<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 490: CUDA_DES_PerformSearch490<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 491: CUDA_DES_PerformSearch491<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 492: CUDA_DES_PerformSearch492<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 493: CUDA_DES_PerformSearch493<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 494: CUDA_DES_PerformSearch494<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 495: CUDA_DES_PerformSearch495<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 496: CUDA_DES_PerformSearch496<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 497: CUDA_DES_PerformSearch497<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 498: CUDA_DES_PerformSearch498<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 499: CUDA_DES_PerformSearch499<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 500: CUDA_DES_PerformSearch500<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 501: CUDA_DES_PerformSearch501<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 502: CUDA_DES_PerformSearch502<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 503: CUDA_DES_PerformSearch503<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 504: CUDA_DES_PerformSearch504<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 505: CUDA_DES_PerformSearch505<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 506: CUDA_DES_PerformSearch506<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 507: CUDA_DES_PerformSearch507<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 508: CUDA_DES_PerformSearch508<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 509: CUDA_DES_PerformSearch509<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 510: CUDA_DES_PerformSearch510<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 511: CUDA_DES_PerformSearch511<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
