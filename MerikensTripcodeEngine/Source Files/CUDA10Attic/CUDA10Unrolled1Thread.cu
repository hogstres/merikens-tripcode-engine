#include "hip/hip_runtime.h"
// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



// TO DO: Use smallKeyBitmap[]!



///////////////////////////////////////////////////////////////////////////////
// INCLUDE FILE(S)                                                           //
///////////////////////////////////////////////////////////////////////////////

#include "MerikensTripcodeEngine.h"



///////////////////////////////////////////////////////////////////////////////
// VARIABLES FOR CUDA CODES                                                  //
///////////////////////////////////////////////////////////////////////////////

__device__ __constant__ unsigned char   CUDA_keyCharTable_OneByte[SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   CUDA_keyCharTable_FirstByte  [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   CUDA_keyCharTable_SecondByte [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ char            CUDA_base64CharTable[64];
__device__ __constant__ unsigned char   CUDA_key[12];



///////////////////////////////////////////////////////////////////////////////
// BITSLICE DES                                                              //
///////////////////////////////////////////////////////////////////////////////

#define CUDA_DES_BS_DEPTH                   32
#define CUDA_DES_NUM_THREADS_PER_BLOCK      128
#define CUDA_DES_NUM_THREADS_FOR_BITSLICE   1
#define CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK (CUDA_DES_NUM_THREADS_PER_BLOCK / CUDA_DES_NUM_THREADS_FOR_BITSLICE)

typedef int           DES_ARCH_WORD;
typedef int           DES_ARCH_WORD_32;
#define DES_ARCH_SIZE 4
#define DES_ARCH_BITS 32

typedef int           DES_Vector;
// #define CUDA_DES_BS_DEPTH  DES_ARCH_BITS
#define DES_VECTOR_ZERO               0
#define DES_VECTOR_ONES               ~(DES_Vector)0

#define DES_VECTOR_NOT(dst, a)        (dst) =  ~(a)
#define DES_VECTOR_AND(dst, a, b)     (dst) =   (a) &  (b)
#define DES_VECTOR_OR(dst, a, b)      (dst) =   (a) |  (b)
#define DES_VECTOR_AND_NOT(dst, a, b) (dst) =   (a) & ~(b)
#define DES_VECTOR_XOR_NOT(dst, a, b) (dst) = ~((a) ^  (b))
#define DES_VECTOR_NOT_OR(dst, a, b)  (dst) = ~((a) |  (b))
#define DES_VECTOR_SEL(dst, a, b, c)  (dst) = (((a) & ~(c)) ^ ((b) & (c)))
#define DES_VECTOR_XOR_FUNC(a, b)              ((a) ^  (b))
#define DES_VECTOR_XOR(dst, a, b)     (dst) = DES_VECTOR_XOR_FUNC((a), (b))
#define DES_VECTOR_SET(dst, ofs, src) *((DES_Vector *)((DES_Vector *)&(dst) + ((ofs) << DB_SHIFT))) = (src)

#define DES_CONSTANT_QUALIFIERS      __device__ __constant__
#define DES_FUNCTION_QUALIFIERS      __device__ __forceinline__
#define DES_SBOX_FUNCTION_QUALIFIERS __device__ __forceinline__

__device__ __shared__ DES_Vector dataBlocks[32 * CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK];
#define DB_SHIFT 4

const unsigned char expansionTable[48] = {
	31,  0,  1,  2,  3,  4,
	 3,  4,  5,  6,  7,  8,
	 7,  8,  9, 10, 11, 12,
	11, 12, 13, 14, 15, 16,
	15, 16, 17, 18, 19, 20,
	19, 20, 21, 22, 23, 24,
	23, 24, 25, 26, 27, 28,
	27, 28, 29, 30, 31,  0
};

__device__ __constant__ unsigned char CUDA_expansionFunction[96];
__device__ __constant__ unsigned char CUDA_key7Array[CUDA_DES_BS_DEPTH];
__device__ __constant__ DES_Vector    CUDA_keyFrom49To55Array[7];

const char charToIndexTableForDES[0x100] = {
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x00, 0x01,
	0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09,
	0x0a, 0x0b, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x0c, 0x0d, 0x0e, 0x0f, 0x10, 0x11, 0x12,
	0x13, 0x14, 0x15, 0x16, 0x17, 0x18, 0x19, 0x1a,
	0x1b, 0x1c, 0x1d, 0x1e, 0x1f, 0x20, 0x21, 0x22,
	0x23, 0x24, 0x25, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x26, 0x27, 0x28, 0x29, 0x2a, 0x2b, 0x2c,
	0x2d, 0x2e, 0x2f, 0x30, 0x31, 0x32, 0x33, 0x34,
	0x35, 0x36, 0x37, 0x38, 0x39, 0x3a, 0x3b, 0x3c,
	0x3d, 0x3e, 0x3f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
};

DES_CONSTANT_QUALIFIERS char CUDA_DES_indexToCharTable[64] =
//	"./0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
{
	/* 00 */ '.', '/',
	/* 02 */ '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 
	/* 12 */ 'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 
	/* 28 */ 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
	/* 38 */ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p',
	/* 54 */ 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z', 
};

DES_CONSTANT_QUALIFIERS unsigned char keySchedule[DES_SIZE_KEY_SCHEDULE] = {
	12, 46, 33, 52, 48, 20, 34, 55,  5, 13, 18, 40,  4, 32, 26, 27,
	38, 54, 53,  6, 31, 25, 19, 41, 15, 24, 28, 43, 30,  3, 35, 22,
	 2, 44, 14, 23, 51, 16, 29, 49,  7, 17, 37,  8,  9, 50, 42, 21,
	 5, 39, 26, 45, 41, 13, 27, 48, 53,  6, 11, 33, 52, 25, 19, 20,
	31, 47, 46, 54, 55, 18, 12, 34,  8, 17, 21, 36, 23, 49, 28, 15,
	24, 37,  7, 16, 44,  9, 22, 42,  0, 10, 30,  1,  2, 43, 35, 14,
	46, 25, 12, 31, 27, 54, 13, 34, 39, 47, 52, 19, 38, 11,  5,  6,
	48, 33, 32, 40, 41,  4, 53, 20, 51,  3,  7, 22,  9, 35, 14,  1,
	10, 23, 50,  2, 30, 24,  8, 28, 43, 49, 16, 44, 17, 29, 21,  0,
	32, 11, 53, 48, 13, 40, 54, 20, 25, 33, 38,  5, 55, 52, 46, 47,
	34, 19, 18, 26, 27, 45, 39,  6, 37, 42, 50,  8, 24, 21,  0, 44,
	49,  9, 36, 17, 16, 10, 51, 14, 29, 35,  2, 30,  3, 15,  7, 43,
	18, 52, 39, 34, 54, 26, 40,  6, 11, 19, 55, 46, 41, 38, 32, 33,
	20,  5,  4, 12, 13, 31, 25, 47, 23, 28, 36, 51, 10,  7, 43, 30,
	35, 24, 22,  3,  2, 49, 37,  0, 15, 21, 17, 16, 42,  1, 50, 29,
	 4, 38, 25, 20, 40, 12, 26, 47, 52,  5, 41, 32, 27, 55, 18, 19,
	 6, 46, 45, 53, 54, 48, 11, 33,  9, 14, 22, 37, 49, 50, 29, 16,
	21, 10,  8, 42, 17, 35, 23, 43,  1,  7,  3,  2, 28, 44, 36, 15,
	45, 55, 11,  6, 26, 53, 12, 33, 38, 46, 27, 18, 13, 41,  4,  5,
	47, 32, 31, 39, 40, 34, 52, 19, 24,  0,  8, 23, 35, 36, 15,  2,
	 7, 49, 51, 28,  3, 21,  9, 29, 44, 50, 42, 17, 14, 30, 22,  1,
	31, 41, 52, 47, 12, 39, 53, 19, 55, 32, 13,  4, 54, 27, 45, 46,
	33, 18, 48, 25, 26, 20, 38,  5, 10, 43, 51,  9, 21, 22,  1, 17,
	50, 35, 37, 14, 42,  7, 24, 15, 30, 36, 28,  3,  0, 16,  8, 44,
	55, 34, 45, 40,  5, 32, 46, 12, 48, 25,  6, 52, 47, 20, 38, 39,
	26, 11, 41, 18, 19, 13, 31, 53,  3, 36, 44,  2, 14, 15, 51, 10,
	43, 28, 30,  7, 35,  0, 17,  8, 23, 29, 21, 49, 50,  9,  1, 37,
	41, 20, 31, 26, 46, 18, 32, 53, 34, 11, 47, 38, 33,  6, 55, 25,
	12, 52, 27,  4,  5, 54, 48, 39, 42, 22, 30, 17,  0,  1, 37, 49,
	29, 14, 16, 50, 21, 43,  3, 51,  9, 15,  7, 35, 36, 24, 44, 23,
	27,  6, 48, 12, 32,  4, 18, 39, 20, 52, 33, 55, 19, 47, 41, 11,
	53, 38, 13, 45, 46, 40, 34, 25, 28,  8, 16,  3, 43, 44, 23, 35,
	15,  0,  2, 36,  7, 29, 42, 37, 24,  1, 50, 21, 22, 10, 30,  9,
	13, 47, 34, 53, 18, 45,  4, 25,  6, 38, 19, 41,  5, 33, 27, 52,
	39, 55, 54, 31, 32, 26, 20, 11, 14, 51,  2, 42, 29, 30,  9, 21,
	 1, 43, 17, 22, 50, 15, 28, 23, 10, 44, 36,  7,  8, 49, 16, 24,
	54, 33, 20, 39,  4, 31, 45, 11, 47, 55,  5, 27, 46, 19, 13, 38,
	25, 41, 40, 48, 18, 12,  6, 52,  0, 37, 17, 28, 15, 16, 24,  7,
	44, 29,  3,  8, 36,  1, 14,  9, 49, 30, 22, 50, 51, 35,  2, 10,
	40, 19,  6, 25, 45, 48, 31, 52, 33, 41, 46, 13, 32,  5, 54, 55,
	11, 27, 26, 34,  4, 53, 47, 38, 43, 23,  3, 14,  1,  2, 10, 50,
	30, 15, 42, 51, 22, 44,  0, 24, 35, 16,  8, 36, 37, 21, 17, 49,
	26,  5, 47, 11, 31, 34, 48, 38, 19, 27, 32, 54, 18, 46, 40, 41,
	52, 13, 12, 20, 45, 39, 33, 55, 29,  9, 42,  0, 44, 17, 49, 36,
	16,  1, 28, 37,  8, 30, 43, 10, 21,  2, 51, 22, 23,  7,  3, 35,
	19, 53, 40,  4, 55, 27, 41, 31, 12, 20, 25, 47, 11, 39, 33, 34,
	45,  6,  5, 13, 38, 32, 26, 48, 22,  2, 35, 50, 37, 10, 42, 29,
	 9, 51, 21, 30,  1, 23, 36,  3, 14, 24, 44, 15, 16,  0, 49, 28,
};

void DES_CreateExpansionFunction(char *saltString, unsigned char *expansionFunction)
{
	unsigned char saltChar1 = '.', saltChar2 = '.';
	DES_ARCH_WORD salt;
	DES_ARCH_WORD mask;
	int src, dst;

	if (saltString[0]) {
		saltChar1 = saltString[0];
		if (saltString[1])
			saltChar2 = saltString[1];
	}
	salt =    charToIndexTableForDES[saltChar1]
	       | (charToIndexTableForDES[saltChar2] << 6);

	mask = 1;
	for (dst = 0; dst < 48; dst++) {
		if (dst == 24) mask = 1;

		if (salt & mask) {
			if (dst < 24) src = dst + 24; else src = dst - 24;
		} else src = dst;

		expansionFunction[dst     ] = expansionTable[src];
		expansionFunction[dst + 48] = expansionTable[src] + 32;

		mask <<= 1;
	}
}

// Bitslice DES S-boxes for x86 with MMX/SSE2/AVX and for typical RISC
// architectures.  These use AND, OR, XOR, NOT, and AND-NOT gates.
//
// Gate counts: 49 44 46 33 48 46 46 41
// Average: 44.125
//
// Several same-gate-count expressions for each S-box are included (for use on
// different CPUs/GPUs).
//
// These Boolean expressions corresponding to DES S-boxes have been generated
// by Roman Rusakov <roman_rus at openwall.com> for use in Openwall's
// John the Ripper password cracker: http://www.openwall.com/john/
// Being mathematical formulas, they are not copyrighted and are free for reuse
// by anyone.
//
// This file (a specific representation of the S-box expressions, surrounding
// logic) is Copyright (c) 2011 by Solar Designer <solar at openwall.com>.
// Redistribution and use in source and binary forms, with or without
// modification, are permitted.  (This is a heavily cut-down "BSD license".)
//
// The effort has been sponsored by Rapid7: http://www.rapid7.com

//
// s1-00484, 49 gates, 17 regs, 11 andn, 4/9/39/79/120 stalls, 74 biop
// Currently used for MMX/SSE2 and x86-64 SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s1(
	DES_Vector arg1,
	DES_Vector arg2,
	DES_Vector arg3,
	DES_Vector arg4,
	DES_Vector arg5,
	DES_Vector arg6,
    volatile DES_Vector *out1,
    volatile DES_Vector *out2,
    volatile DES_Vector *out3,
    volatile DES_Vector *out4
) {
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"
	    ".reg .u32 t11;         \n\t"
	    ".reg .u32 t12;         \n\t"
	    ".reg .u32 t13;         \n\t"
	    
	    "not.b32 t0,  %8;      \n\t"
	    "and.b32 t0,  %4, t0;  \n\t"
	    "xor.b32 t1,  %7, t0;  \n\t"
	    "or.b32  t2,  %6, %9; \n\t"
	    "xor.b32 t3,  %4, %6; \n\t"
	    "and.b32 t4,  t2,  t3;  \n\t"
	    "xor.b32 t5,  %7, t4;  \n\t"
	    "not.b32 t6,  t1;       \n\t"
	    "and.b32 t6,  t5,  t6;  \n\t"

	    "xor.b32 t7,  %8, %9; \n\t"
	    "xor.b32 t8,  %6, t7;  \n\t"
	    "not.b32 t9,  t8;       \n\t"
	    "and.b32 t9,  t1,  t9;  \n\t"
	    "or.b32  t8,  %9, t4;  \n\t"
	    "xor.b32 t4,  t9,  t8;  \n\t"
	    "not.b32 t8,  t6;       \n\t"
	    "and.b32 t8,  t4,  t8;  \n\t"

	    "or.b32  t9,  %4, %9; \n\t"
	    "or.b32  t10, t4,  t9;  \n\t"
	    "not.b32 t11, t5;       \n\t"
	    "and.b32 t11, %8, t11; \n\t"
	    "xor.b32 t5,  t10, t11; \n\t"

	    "not.b32 t12, t9;       \n\t"
		"and.b32 t12, %7, t12; \n\t"
	    "xor.b32 t9,  t11, t12; \n\t"
	    "not.b32 t12, t3;       \n\t"
	    "and.b32 t12, t7,  t12; \n\t"
	    "or.b32  t3,  t9,  t12; \n\t"

	    "not.b32 t12, t0;       \n\t"
	    "and.b32 t12, %6, t12; \n\t"
	    "xor.b32 t0,  t1,  t10; \n\t"
	    "not.b32 t9,  t12;      \n\t"
	    "and.b32 t9,  t0,  t9;  \n\t"
	    "not.b32 t12, t9;       \n\t"
	    "and.b32 t0,  t2,  t4;  \n\t"
	    "xor.b32 t4,  t12, t0;  \n\t"
	    "not.b32 t13, %5;      \n\t"
	    "and.b32 t13, t5,  t13; \n\t"
	    "xor.b32 t5,  t13, t4;  \n\t"
	    "xor.b32 %2, %2, t5;  \n\t"
	
	    "xor.b32 t12, t7,  t9;  \n\t"
	    "or.b32  t0,  t11, t12; \n\t"
	    "xor.b32 t5,  t2,  t0;  \n\t"
	    "xor.b32 t11, %4, t5;  \n\t"
	    "xor.b32 t5,  t4,  t11; \n\t"
	    "or.b32  t9,  t6,  %5; \n\t"
	    "xor.b32 t12, t9,  t5;  \n\t"
	    "xor.b32 %0, %0, t12; \n\t"
	
	    "xor.b32 t13, t2,  t10; \n\t"
	    "or.b32  t0,  t3,  t13; \n\t"
	    "xor.b32 t13, t11, t0;  \n\t"
	    "or.b32  t0,  t7,  t5;  \n\t"
	    "xor.b32 t5,  t13, t0;  \n\t"
	    "or.b32  t0,  t8,  %5; \n\t"
	    "xor.b32 t6,  t0,  t5;  \n\t"
	    "xor.b32 %1, %1, t6;  \n\t"

	    "or.b32  t6,  %8, t1;  \n\t"
	    "not.b32 t9,  t13;      \n\t"
	    "and.b32 t9,  t6,  t9;  \n\t"
	    "and.b32 t13, t8,  t11; \n\t"
	    "xor.b32 t11, t9,  t13; \n\t"
	    "or.b32  t13, t11, %5; \n\t"
	    "xor.b32 t12, t13, t3;  \n\t"
	    "xor.b32 %3, %3, t12;   \n\t"
	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(arg1)     // %4
	      "r"(arg2)     // %5
	      "r"(arg3)     // %6
	      "r"(arg4)     // %7
	      "r"(arg5)     // %8
	      "r"(arg6));   // %9
}

//
// s2-016251, 44 gates, 14 regs, 13 andn, 1/9/22/61/108 stalls, 66 biop */
//
DES_SBOX_FUNCTION_QUALIFIERS void
s2(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"
	    ".reg .u32 t11;         \n\t"
	    ".reg .u32 t12;         \n\t"

		"xor.b32 t0, %5, %8;    \n\t"

		"not.b32 t1, %9;        \n\t"
		"and.b32 t1, %4, t1;    \n\t"
		"not.b32 t2, t1;        \n\t"
		"and.b32 t2, %8, t2;    \n\t"
		"or.b32  t1, %5, t2;    \n\t"

		"not.b32 t3, %9;        \n\t"
		"and.b32 t3, t0, t3;    \n\t"
		"and.b32 t4, %4, t0;    \n\t"
		"xor.b32 t5, %8, t4;    \n\t"
		"not.b32 t6, t3;        \n\t"
		"and.b32 t6, t5, t6;    \n\t"

		"and.b32 t7, %6, %9;    \n\t"
		"xor.b32 t8, t2, t3;    \n\t"
		"and.b32 t2, t1, t8;    \n\t"
		"not.b32 t3, t7;        \n\t"
		"and.b32 t3, t2, t3;    \n\t"

		"and.b32 t8, %6, t2;    \n\t"
		"not.b32 t2, %4;        \n\t"
		"xor.b32 t9, t8, t2;    \n\t"
		"xor.b32 t2, %9, t0;    \n\t"
		"not.b32 t0, t7;        \n\t"
		"and.b32 t0, t2, t0;    \n\t"
		"xor.b32 t10, t9, t0;   \n\t"
		"not.b32 t11, t3;       \n\t"
		"and.b32 t11, %7, t11;  \n\t"
		"xor.b32 t3, t11, t10;  \n\t"
		"xor.b32 %1, %1, t3;    \n\t"

		"not.b32 t3, t0;        \n\t"
		"and.b32 t3, %5, t3;    \n\t"
		"xor.b32 t0, t5, t3;    \n\t"
		"not.b32 t5, t0;        \n\t"
		"and.b32 t5, t9, t5;    \n\t"
		"xor.b32 t9, %6, t2;    \n\t"
		"xor.b32 t11, t5, t9;   \n\t"
		"not.b32 t5, %7;        \n\t"
		"and.b32 t5, t1, t5;    \n\t"
		"xor.b32 t12, t5, t11;  \n\t"
		"xor.b32 %0, %0, t12;   \n\t"

		"xor.b32 t5, t8, t3;    \n\t"
		"or.b32  t3, t9, t5;    \n\t"
		"xor.b32 t8, t1, t10;   \n\t"
		"or.b32  t1, t7, t8;    \n\t"
		"xor.b32 t7, t3, t1;    \n\t"

		"not.b32 t1, t11;       \n\t"
		"and.b32 t1, t10, t1;   \n\t"
		"xor.b32 t3, t4, t5;    \n\t"
		"or.b32  t4, t1, t3;    \n\t"
		"not.b32 t1, t9;        \n\t"
		"and.b32 t1, t6, t1;    \n\t"
		"xor.b32 t3, t4, t1;    \n\t"
		"or.b32  t1, t3, %7;    \n\t"
		"xor.b32 t4, t1, t7;    \n\t"
		"xor.b32 %2, %2, t4;    \n\t"

		"not.b32 t1, t0;        \n\t"
		"and.b32 t1, t3, t1;    \n\t"
		"or.b32  t0, t2, t8;    \n\t"
		"xor.b32 t2, t1, t0;    \n\t"
		"or.b32  t0, t6, %7;    \n\t"
		"xor.b32 t1, t0, t2;    \n\t"
		"xor.b32 %3, %3, t1;    \n\t"
		
		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s3-000426, 46 gates, 16 regs, 14 andn, 2/5/12/35/75 stalls, 68 biop
// Currently used for x86-64 SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s3(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"

		"not.b32 t0, %5;        \n\t"
		"and.b32 t0, %4, t0;    \n\t"
		"xor.b32 t1, %6, %9;    \n\t"
		"or.b32  t2, t0, t1;    \n\t"
		"xor.b32 t0, %7, %9;    \n\t"
		"not.b32 t3, %4;        \n\t"
		"and.b32 t3, t0, t3;    \n\t"
		"xor.b32 t4, t2, t3;    \n\t"

		"xor.b32 t5, %5, t1;    \n\t"
		"not.b32 t6, %9;        \n\t"
		"and.b32 t6, t5, t6;    \n\t"
		"xor.b32 t7, t2, t6;    \n\t"
		"not.b32 t2, t7;        \n\t"
		"and.b32 t2, t4, t2;    \n\t"

		"and.b32 t6, %9, t4;    \n\t"
		"or.b32  t8, %7, t6;    \n\t"
		"and.b32 t6, %4, t8;    \n\t"
		"xor.b32 t8, t5, t6;    \n\t"
		"not.b32 t6, %8;        \n\t"
		"and.b32 t6, t4, t6;    \n\t"
		"xor.b32 t9, t6, t8;    \n\t"
		"xor.b32 %3, %3, t9;    \n\t"

		"and.b32 t6, t1, t0;    \n\t"
		"xor.b32 t0, %4, %7;    \n\t"
		"xor.b32 t9, t7, t0;    \n\t"
		"or.b32  t7, %6, t9;    \n\t"
		"not.b32 t9, t6;        \n\t"
		"and.b32 t9, t7, t9;    \n\t"

		"or.b32  t6, t3, t0;    \n\t"
		"not.b32 t0, t6;        \n\t"
		"and.b32 t0, t8, t0;    \n\t"
		"and.b32 t7, %7, %9;    \n\t"
		"not.b32 t8, %5;        \n\t"
		"and.b32 t8, t7, t8;    \n\t"
		"xor.b32 t10, t0, t8;   \n\t"

		"not.b32 t0, %6;        \n\t"
		"and.b32 t0, t10, t0;   \n\t"
		"or.b32  t8, t5, t7;    \n\t"
		"not.b32 t7, t0;        \n\t"
		"and.b32 t7, t8, t7;    \n\t"
		"xor.b32 t0, %4, t7;    \n\t"
		"and.b32 t7, t9, %8;    \n\t"
		"xor.b32 t8, t7, t0;    \n\t"
		"xor.b32 %1, %1, t8;    \n\t"

		"not.b32 t0, %5;        \n\t"
		"and.b32 t0, t4, t0;    \n\t"
		"not.b32 t4, %6;        \n\t"
		"and.b32 t4, t0, t4;    \n\t"
		"xor.b32 t7, t5, t6;    \n\t"
		"not.b32 t6, t7;        \n\t"
		"xor.b32 t7, t4, t6;    \n\t"
		"not.b32 t4, t2;        \n\t"
		"and.b32 t4, %8, t4;    \n\t"
		"xor.b32 t2, t4, t7;    \n\t"
		"xor.b32 %0, %0, t2;    \n\t"

		"and.b32 t2, %7, t1;    \n\t"
		"or.b32  t1, t5, t7;    \n\t"
		"not.b32 t4, t2;        \n\t"
		"and.b32 t4, t1, t4;    \n\t"
		"or.b32  t1, t3, t0;    \n\t"
		"xor.b32 t0, t4, t1;    \n\t"
		"or.b32  t1, t10, %8;   \n\t"
		"xor.b32 t2, t1, t0;    \n\t"
		"xor.b32 %2, %2, t2;    \n\t"
		
		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s4, 33 gates, 11/12 regs, 9 andn, 2/21/53/86/119 stalls, 52 biop
//
DES_SBOX_FUNCTION_QUALIFIERS void
s4(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"

	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	
		"xor.b32 t0, %4, %6;    \n\t"
		"xor.b32 t1, %6, %8;    \n\t"
		"or.b32  t2, %5, %7;    \n\t"
		"xor.b32 t3, %8, t2;    \n\t"
		"not.b32 t2, t3;        \n\t"
		"and.b32 t2, t1, t2;    \n\t"
		"not.b32 t3, %5;        \n\t"
		"and.b32 t3, t1, t3;    \n\t"
		"xor.b32 t4, %7, t3;    \n\t"
		"or.b32  t5, t0, t4;    \n\t"
		"not.b32 t6, t2;        \n\t"
		"and.b32 t6, t5, t6;    \n\t"
		"xor.b32 t5, %5, t6;    \n\t"

		"and.b32 t7, t4, t5;    \n\t"
		"not.b32 t4, t7;        \n\t"
		"and.b32 t4, t1, t4;    \n\t"
		"xor.b32 t1, t0, t5;    \n\t"
		"not.b32 t0, t4;        \n\t"
		"and.b32 t0, t1, t0;    \n\t"
		"xor.b32 t4, t2, t0;    \n\t"

		"xor.b32 t0, %5, %7;    \n\t"
		"or.b32  t2, %8, t3;    \n\t"
		"xor.b32 t3, t1, t2;    \n\t"
		"not.b32 t1, t0;        \n\t"
		"and.b32 t1, t3, t1;    \n\t"
		"xor.b32 t2, t6, t1;    \n\t"
		"not.b32 t1, t4;        \n\t"
		"and.b32 t1, %9, t1;    \n\t"
		"xor.b32 t6, t1, t2;    \n\t"
		"xor.b32 %0, %0, t6;    \n\t"

		"not.b32 t1, t2;        \n\t"
		"not.b32 t2, %9;        \n\t"
		"and.b32 t2, t4, t2;    \n\t"
		"xor.b32 t6, t2, t1;    \n\t"
		"xor.b32 %1, %1, t6;    \n\t"

		"xor.b32 t2, t4, t1;    \n\t"
		"not.b32 t1, t0;        \n\t"
		"and.b32 t1, t2, t1;    \n\t"
		"or.b32  t0, t7, t1;    \n\t"
		"xor.b32 t1, t3, t0;    \n\t"
		"or.b32  t0, t5, %9;    \n\t"
		"xor.b32 t2, t0, t1;    \n\t"
		"xor.b32 %2, %2, t2;    \n\t"

		"and.b32 t0, %9, t5;    \n\t"
		"xor.b32 t2, t0, t1;    \n\t"
		"xor.b32 %3, %3, t2;    \n\t"
		
		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s5-04832, 48 gates, 15/16 regs, 9 andn, 5/23/62/109/159 stalls, 72 biop
// Currently used for MMX/SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s5(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"

	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;          \n\t"
	    ".reg .u32 t11;          \n\t"
	    ".reg .u32 t12;          \n\t"
	
		"or.b32 t1, %4, %6; \n\t"
		"not.b32 t10, %9; \n\t"
		"and.b32 t10, t1, t10; \n\t"
		"xor.b32 t6, %4, t10; \n\t"
		"xor.b32 t2, %6, t6; \n\t"
		"or.b32 t3, %7, t2; \n\t"
		"not.b32 t7, %7; \n\t"
		"and.b32 t7, t10, t7; \n\t"
		"xor.b32 t10, %6, t7; \n\t"
		"and.b32 t7, %8, t10; \n\t"
		"or.b32 t12, %4, t2; \n\t"
		"xor.b32 t2, t7, t12; \n\t"
		"xor.b32 t7, %7, t2; \n\t"
		"xor.b32 t2, %9, t7; \n\t"
		"or.b32 t4, t6, t2; \n\t"
		"and.b32 t8, %8, t4; \n\t"
		"xor.b32 t11, t6, t8; \n\t"
		"and.b32 t9, %7, t12; \n\t"
		"xor.b32 t5, t11, t9; \n\t"
		"not.b32 t11, %4; \n\t"
		"and.b32 t11, t4, t11; \n\t"
		"xor.b32 t4, t10, t11; \n\t"
		"xor.b32 t9, %8, t3; \n\t"
		"not.b32 t0, t4; \n\t"
		"and.b32 t0, t9, t0; \n\t"
		"not.b32 t4, t0; \n\t"
		"not.b32 t0, %5; \n\t"
		"and.b32 t0, t4, t0; \n\t"
		"xor.b32 t4, t0, t7; \n\t"
		"xor.b32 %2, %2, t4; \n\t"
		"not.b32 t7, t8; \n\t"
		"and.b32 t7, t10, t7; \n\t"
		"xor.b32 t0, t11, t9; \n\t"
		"or.b32 t11, t5, t0; \n\t"
		"not.b32 t4, t7; \n\t"
		"and.b32 t4, t11, t4; \n\t"
		"not.b32 t0, t4; \n\t"
		"and.b32 t0, t3, t0; \n\t"
		"and.b32 t11, t2, t4; \n\t"
		"xor.b32 t7, t9, t11; \n\t"
		"and.b32 t2, t10, t12; \n\t"
		"or.b32 t11, t7, t2; \n\t"
		"xor.b32 t9, t8, t11; \n\t"
		"and.b32 t11, t9, %5; \n\t"
		"xor.b32 t12, t11, t5; \n\t"
		"xor.b32 %3, %3, t12; \n\t"
		"xor.b32 t12, t1, t4; \n\t"
		"xor.b32 t2, %4, t12; \n\t"
		"and.b32 t11, %7, t7; \n\t"
		"xor.b32 t8, t2, t11; \n\t"
		"or.b32 t12, t0, %5; \n\t"
		"xor.b32 t11, t12, t8; \n\t"
		"xor.b32 %0, %0, t11; \n\t"
		"xor.b32 t9, t3, t10; \n\t"
		"not.b32 t5, t8; \n\t"
		"and.b32 t5, t9, t5; \n\t"
		"xor.b32 t4, t6, t7; \n\t"
		"xor.b32 t1, t5, t4; \n\t"
		"and.b32 t2, t3, %5; \n\t"
		"xor.b32 t0, t2, t1; \n\t"
		"xor.b32 %1, %1, t0; \n\t"

		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s6-000007, 46 gates, 19 regs, 8 andn, 3/19/39/66/101 stalls, 69 biop
// Currently used for x86-64 SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s6(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"
	    ".reg .u32 t11;         \n\t"
	    ".reg .u32 t12;         \n\t"
	    ".reg .u32 t13;         \n\t"
	    
		"xor.b32 t0, %5, %8; \n\t"

		"or.b32 t8, %5, %9; \n\t"
		"and.b32 t1, %4, t8; \n\t"
		"xor.b32 t8, t0, t1; \n\t"
		"xor.b32 t0, %9, t8; \n\t"
		"not.b32 t12, t0; \n\t"
		"and.b32 t12, %8, t12; \n\t"

		"and.b32 t11, %4, t0; \n\t"
		"xor.b32 t0, %5, t11; \n\t"
		"xor.b32 t4, %4, %6; \n\t"
		"or.b32 t13, t0, t4; \n\t"
		"xor.b32 t2, t8, t13; \n\t"

		"and.b32 t7, %6, t2; \n\t"
		"not.b32 t6, %9; \n\t"
		"and.b32 t6, t7, t6; \n\t"
		"or.b32 t9, t12, t0; \n\t"
		"xor.b32 t0, t6, t9; \n\t"
		"and.b32 t10, t0, %7; \n\t"
		"xor.b32 t5, t10, t2; \n\t"
		"xor.b32 %3, %3, t5; \n\t"

		"xor.b32 t5, %5, t13; \n\t"
		"not.b32 t13, t5; \n\t"
		"and.b32 t13, %9, t13; \n\t"
		"xor.b32 t10, %6, t13; \n\t"
		"not.b32 t13, t7; \n\t"
		"and.b32 t13, %8, t13; \n\t"
		"or.b32 t3, t10, t13; \n\t"

		"or.b32 t13, %4, t2; \n\t"
		"and.b32 t2, t9, t13; \n\t"
		"xor.b32 t9, t10, t2; \n\t"
		"not.b32 t13, t6; \n\t"
		"and.b32 t13, t9, t13; \n\t"
		"or.b32 t6, t12, %7; \n\t"
		"xor.b32 t12, t6, t13; \n\t"
		"xor.b32 %2, %2, t12; \n\t"

		"or.b32 t2, %5, t4; \n\t"
		"xor.b32 t6, t0, t2; \n\t"
		"or.b32 t12, t1, t3; \n\t"
		"xor.b32 t13, t6, t12; \n\t"

		"xor.b32 t4, t8, t9; \n\t"
		"not.b32 t0, t4; \n\t"
		"and.b32 t0, %8, t0; \n\t"
		"not.b32 t1, t5; \n\t"
		"xor.b32 t6, t2, t1; \n\t"
		"xor.b32 t12, t0, t6; \n\t"
		"not.b32 t9, %7; \n\t"
		"and.b32 t9, t12, t9; \n\t"
		"xor.b32 t12, t9, t13; \n\t"
		"xor.b32 %1, %1, t12; \n\t"

		"xor.b32 t9, %9, t11; \n\t"
		"xor.b32 t8, %4, t10; \n\t"
		"and.b32 t4, t9, t8; \n\t"
		"xor.b32 t5, t7, t6; \n\t"
		"xor.b32 t2, t4, t5; \n\t"
		"not.b32 t1, %7; \n\t"
		"and.b32 t1, t3, t1; \n\t"
		"xor.b32 t0, t1, t2; \n\t"
		"xor.b32 %0, %0, t0; \n\t"

	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(a1)     // %4
	      "r"(a2)     // %5
	      "r"(a3)     // %6
	      "r"(a4)     // %7
	      "r"(a5)     // %8
	      "r"(a6));   // %9
}

//
// s7-056945, 46 gates, 16 regs, 7 andn, 10/31/62/107/156 stalls, 67 biop
// Currently used for MMX/SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s7(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    
		"xor.b32 t6, %7, %8; \n\t"
		"xor.b32 t3, %6, t6; \n\t"
		"and.b32 t1, %9, t3; \n\t"
		"and.b32 t2, %7, t6; \n\t"
		"xor.b32 t4, %5, t2; \n\t"
		"and.b32 t0, t1, t4; \n\t"

		"and.b32 t7, %9, t2; \n\t"
		"xor.b32 t5, %6, t7; \n\t"
		"or.b32 t7, t4, t5; \n\t"
		"xor.b32 t8, %9, t6; \n\t"
		"xor.b32 t6, t7, t8; \n\t"
		"not.b32 t7, t0; \n\t"
		"and.b32 t7, %4, t7; \n\t"
		"xor.b32 t9, t7, t6; \n\t"
		"xor.b32 %3, %3, t9; \n\t"

		"not.b32 t7, t3; \n\t"
		"and.b32 t7, %8, t7; \n\t"
		"or.b32 t0, t4, t7; \n\t"
		"xor.b32 t9, t1, t5; \n\t"
		"xor.b32 t5, t0, t9; \n\t"

		"xor.b32 t0, t1, t8; \n\t"
		"not.b32 t1, t0; \n\t"
		"and.b32 t1, %7, t1; \n\t"
		"not.b32 t0, t1; \n\t"
		"and.b32 t0, t4, t0; \n\t"
		"xor.b32 t4, %8, t9; \n\t"
		"xor.b32 t1, t0, t4; \n\t"

		"or.b32 t9, t2, t6; \n\t"
		"and.b32 t0, %6, t1; \n\t"
		"or.b32 t4, t9, t0; \n\t"
		"not.b32 t2, t8; \n\t"
		"and.b32 t2, t3, t2; \n\t"
		"xor.b32 t6, t4, t2; \n\t"
		"not.b32 t8, %4; \n\t"
		"and.b32 t8, t6, t8; \n\t"
		"xor.b32 t9, t8, t5; \n\t"
		"xor.b32 %0, %0, t9; \n\t"

		"or.b32 t9, t1, t6; \n\t"
		"and.b32 t8, %9, t9; \n\t"
		"and.b32 t3, %5, t8; \n\t"
		"xor.b32 t4, t5, t6; \n\t"
		"xor.b32 t2, t3, t4; \n\t"

		"or.b32 t9, t0, t2; \n\t"
		"xor.b32 t5, t8, t9; \n\t"
		"xor.b32 t3, %8, t4; \n\t"
		"or.b32 t0, t5, t3; \n\t"
		"and.b32 t9, t0, %4; \n\t"
		"xor.b32 t5, t9, t1; \n\t"
		"xor.b32 %2, %2, t5; \n\t"

		"xor.b32 t9, t8, t0; \n\t"
		"or.b32 t4, t7, t9; \n\t"
		"not.b32 t5, t6; \n\t"
		"xor.b32 t3, t4, t5; \n\t"
		"not.b32 t1, %4; \n\t"
		"and.b32 t1, t3, t1; \n\t"
		"xor.b32 t0, t1, t2; \n\t"
		"xor.b32 %1, %1, t0; \n\t"


	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(a1)     // %4
	      "r"(a2)     // %5
	      "r"(a3)     // %6
	      "r"(a4)     // %7
	      "r"(a5)     // %8
	      "r"(a6));   // %9
}

//
// s8-004798, 41 gates, 14 regs, 7 andn, 7/35/76/118/160 stalls, 59 biop
// Currently used for MMX/SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s8(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    volatile DES_Vector * out1, volatile DES_Vector * out2, volatile DES_Vector * out3, volatile DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    
		"not.b32 t8, %5; \n\t"
		"and.b32 t8, %6, t8; \n\t"
		"not.b32 t1, %6; \n\t"
		"and.b32 t1, %8, t1; \n\t"
		"xor.b32 t6, %7, t1; \n\t"
		"and.b32 t1, %4, t6; \n\t"
		"not.b32 t7, t8; \n\t"
		"and.b32 t7, t1, t7; \n\t"

		"not.b32 t3, t6; \n\t"
		"and.b32 t3, %5, t3; \n\t"
		"or.b32 t9, %4, t3; \n\t"
		"not.b32 t0, %6; \n\t"
		"and.b32 t0, %5, t0; \n\t"
		"xor.b32 t4, %8, t0; \n\t"
		"and.b32 t0, t9, t4; \n\t"
		"or.b32 t2, t1, t0; \n\t"

		"xor.b32 t1, t6, t0; \n\t"
		"not.b32 t0, t1; \n\t"
		"not.b32 t6, t9; \n\t"
		"and.b32 t6, %6, t6; \n\t"
		"xor.b32 t1, t0, t6; \n\t"
		"xor.b32 t9, t8, t1; \n\t"
		"or.b32 t8, t7, %9; \n\t"
		"xor.b32 t6, t8, t9; \n\t"
		"xor.b32 %1, %1, t6; \n\t"

		"xor.b32 t0, %4, t9; \n\t"
		"and.b32 t6, %8, t0; \n\t"
		"xor.b32 t8, %5, t1; \n\t"
		"xor.b32 t9, t6, t8; \n\t"
		"xor.b32 t1, t3, t9; \n\t"

		"or.b32 t6, %7, t8; \n\t"
		"xor.b32 t3, t1, t6; \n\t"
		"xor.b32 t5, t4, t3; \n\t"
		"xor.b32 t8, %4, t5; \n\t"
		"and.b32 t6, t8, %9; \n\t"
		"xor.b32 t4, t6, t1; \n\t"
		"xor.b32 %3, %3, t4; \n\t"

		"xor.b32 t6, t2, t9; \n\t"
		"or.b32 t4, %5, t6; \n\t"
		"xor.b32 t3, t0, t4; \n\t"
		"xor.b32 t8, %8, t3; \n\t"
		"and.b32 t9, t2, %9; \n\t"
		"xor.b32 t6, t9, t8; \n\t"
		"xor.b32 %2, %2, t6; \n\t"

		"or.b32  t9, %7, t0; \n\t"
		"not.b32 t6, t9; \n\t"
		"and.b32 t6, t8, t6; \n\t"
		"or.b32  t4, t7, t6; \n\t"
		"xor.b32 t3, t5, t4; \n\t"
		"or.b32  t2, t3, %9; \n\t"
		"xor.b32 t0, t2, t1; \n\t"
		"xor.b32 %0, %0, t0; \n\t"

	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(a1)     // %4
	      "r"(a2)     // %5
	      "r"(a3)     // %6
	      "r"(a4)     // %7
	      "r"(a5)     // %8
	      "r"(a6));   // %9
}

#define CLEAR_BLOCK_8(i)                                                             \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 0, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 1, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 2, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 3, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 4, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 5, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 6, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i<<DB_SHIFT)] , 7, DES_VECTOR_ZERO); \

#define CLEAR_BLOCK()  \
	CLEAR_BLOCK_8(0);  \
	CLEAR_BLOCK_8(8);  \
	CLEAR_BLOCK_8(16); \
	CLEAR_BLOCK_8(24); \
	CLEAR_BLOCK_8(32); \
	CLEAR_BLOCK_8(40); \
	CLEAR_BLOCK_8(48); \
	CLEAR_BLOCK_8(56); \

#define w(p, q)    DES_VECTOR_XOR_FUNC(dataBlocks[p], DESContextArray[threadIdx.x].keys[q])
#define x(p)    DES_VECTOR_XOR_FUNC(dataBlocks[CUDA_expansionFunction[p]], DESContextArray[threadIdx.x].keys[keySchedule[keyScheduleIndexBase + (p)]])
#define y(p, q) DES_VECTOR_XOR_FUNC(dataBlocks[p],                         DESContextArray[threadIdx.x].keys[keySchedule[keyScheduleIndexBase + (q)]])
#define z(r)    (&dataBlocks[r])

#define EF(p) (CUDA_expansionFunction[p]<<DB_SHIFT)

DES_FUNCTION_QUALIFIERS
void DES_Crypt(volatile unsigned int keyFrom00To27, volatile unsigned int keyFrom28To48, int intSalt)
{
	if (threadIdx.y == 0)
		CLEAR_BLOCK();
	
	volatile DES_Vector *db = dataBlocks + threadIdx.x;
	
#define K00 ((keyFrom00To27 & (0x1U << 0)) ? 0xffffffffU : 0x0)
#define K01 ((keyFrom00To27 & (0x1U << 1)) ? 0xffffffffU : 0x0)
#define K02 ((keyFrom00To27 & (0x1U << 2)) ? 0xffffffffU : 0x0)
#define K03 ((keyFrom00To27 & (0x1U << 3)) ? 0xffffffffU : 0x0)
#define K04 ((keyFrom00To27 & (0x1U << 4)) ? 0xffffffffU : 0x0)
#define K05 ((keyFrom00To27 & (0x1U << 5)) ? 0xffffffffU : 0x0)
#define K06 ((keyFrom00To27 & (0x1U << 6)) ? 0xffffffffU : 0x0)
#define K07 ((keyFrom00To27 & (0x1U << 7)) ? 0xffffffffU : 0x0)
#define K08 ((keyFrom00To27 & (0x1U << 8)) ? 0xffffffffU : 0x0)
#define K09 ((keyFrom00To27 & (0x1U << 9)) ? 0xffffffffU : 0x0)
#define K10 ((keyFrom00To27 & (0x1U << 10)) ? 0xffffffffU : 0x0)
#define K11 ((keyFrom00To27 & (0x1U << 11)) ? 0xffffffffU : 0x0)
#define K12 ((keyFrom00To27 & (0x1U << 12)) ? 0xffffffffU : 0x0)
#define K13 ((keyFrom00To27 & (0x1U << 13)) ? 0xffffffffU : 0x0)
#define K14 ((keyFrom00To27 & (0x1U << 14)) ? 0xffffffffU : 0x0)
#define K15 ((keyFrom00To27 & (0x1U << 15)) ? 0xffffffffU : 0x0)
#define K16 ((keyFrom00To27 & (0x1U << 16)) ? 0xffffffffU : 0x0)
#define K17 ((keyFrom00To27 & (0x1U << 17)) ? 0xffffffffU : 0x0)
#define K18 ((keyFrom00To27 & (0x1U << 18)) ? 0xffffffffU : 0x0)
#define K19 ((keyFrom00To27 & (0x1U << 19)) ? 0xffffffffU : 0x0)
#define K20 ((keyFrom00To27 & (0x1U << 20)) ? 0xffffffffU : 0x0)
#define K21 ((keyFrom00To27 & (0x1U << 21)) ? 0xffffffffU : 0x0)
#define K22 ((keyFrom00To27 & (0x1U << 22)) ? 0xffffffffU : 0x0)
#define K23 ((keyFrom00To27 & (0x1U << 23)) ? 0xffffffffU : 0x0)
#define K24 ((keyFrom00To27 & (0x1U << 24)) ? 0xffffffffU : 0x0)
#define K25 ((keyFrom00To27 & (0x1U << 25)) ? 0xffffffffU : 0x0)
#define K26 ((keyFrom00To27 & (0x1U << 26)) ? 0xffffffffU : 0x0)
#define K27 ((keyFrom00To27 & (0x1U << 27)) ? 0xffffffffU : 0x0)
#define K28 ((keyFrom28To48 & (0x1U << (28 - 28))) ? 0xffffffffU : 0x0)
#define K29 ((keyFrom28To48 & (0x1U << (29 - 28))) ? 0xffffffffU : 0x0)
#define K30 ((keyFrom28To48 & (0x1U << (30 - 28))) ? 0xffffffffU : 0x0)
#define K31 ((keyFrom28To48 & (0x1U << (31 - 28))) ? 0xffffffffU : 0x0)
#define K32 ((keyFrom28To48 & (0x1U << (32 - 28))) ? 0xffffffffU : 0x0)
#define K33 ((keyFrom28To48 & (0x1U << (33 - 28))) ? 0xffffffffU : 0x0)
#define K34 ((keyFrom28To48 & (0x1U << (34 - 28))) ? 0xffffffffU : 0x0)
#define K35 ((keyFrom28To48 & (0x1U << (35 - 28))) ? 0xffffffffU : 0x0)
#define K36 ((keyFrom28To48 & (0x1U << (36 - 28))) ? 0xffffffffU : 0x0)
#define K37 ((keyFrom28To48 & (0x1U << (37 - 28))) ? 0xffffffffU : 0x0)
#define K38 ((keyFrom28To48 & (0x1U << (38 - 28))) ? 0xffffffffU : 0x0)
#define K39 ((keyFrom28To48 & (0x1U << (39 - 28))) ? 0xffffffffU : 0x0)
#define K40 ((keyFrom28To48 & (0x1U << (40 - 28))) ? 0xffffffffU : 0x0)
#define K41 ((keyFrom28To48 & (0x1U << (41 - 28))) ? 0xffffffffU : 0x0)
#define K42 ((keyFrom28To48 & (0x1U << (42 - 28))) ? 0xffffffffU : 0x0)
#define K43 ((keyFrom28To48 & (0x1U << (43 - 28))) ? 0xffffffffU : 0x0)
#define K44 ((keyFrom28To48 & (0x1U << (44 - 28))) ? 0xffffffffU : 0x0)
#define K45 ((keyFrom28To48 & (0x1U << (45 - 28))) ? 0xffffffffU : 0x0)
#define K46 ((keyFrom28To48 & (0x1U << (46 - 28))) ? 0xffffffffU : 0x0)
#define K47 ((keyFrom28To48 & (0x1U << (47 - 28))) ? 0xffffffffU : 0x0)
#define K48 ((keyFrom28To48 & (0x1U << (48 - 28))) ? 0xffffffffU : 0x0)

#define K00XOR(val) ((keyFrom00To27 & (0x1U << 0)) ? ~(val) : (val))
#define K01XOR(val) ((keyFrom00To27 & (0x1U << 1)) ? ~(val) : (val))
#define K02XOR(val) ((keyFrom00To27 & (0x1U << 2)) ? ~(val) : (val))
#define K03XOR(val) ((keyFrom00To27 & (0x1U << 3)) ? ~(val) : (val))
#define K04XOR(val) ((keyFrom00To27 & (0x1U << 4)) ? ~(val) : (val))
#define K05XOR(val) ((keyFrom00To27 & (0x1U << 5)) ? ~(val) : (val))
#define K06XOR(val) ((keyFrom00To27 & (0x1U << 6)) ? ~(val) : (val))
#define K07XOR(val) ((keyFrom00To27 & (0x1U << 7)) ? ~(val) : (val))
#define K08XOR(val) ((keyFrom00To27 & (0x1U << 8)) ? ~(val) : (val))
#define K09XOR(val) ((keyFrom00To27 & (0x1U << 9)) ? ~(val) : (val))
#define K10XOR(val) ((keyFrom00To27 & (0x1U << 10)) ? ~(val) : (val))
#define K11XOR(val) ((keyFrom00To27 & (0x1U << 11)) ? ~(val) : (val))
#define K12XOR(val) ((keyFrom00To27 & (0x1U << 12)) ? ~(val) : (val))
#define K13XOR(val) ((keyFrom00To27 & (0x1U << 13)) ? ~(val) : (val))
#define K14XOR(val) ((keyFrom00To27 & (0x1U << 14)) ? ~(val) : (val))
#define K15XOR(val) ((keyFrom00To27 & (0x1U << 15)) ? ~(val) : (val))
#define K16XOR(val) ((keyFrom00To27 & (0x1U << 16)) ? ~(val) : (val))
#define K17XOR(val) ((keyFrom00To27 & (0x1U << 17)) ? ~(val) : (val))
#define K18XOR(val) ((keyFrom00To27 & (0x1U << 18)) ? ~(val) : (val))
#define K19XOR(val) ((keyFrom00To27 & (0x1U << 19)) ? ~(val) : (val))
#define K20XOR(val) ((keyFrom00To27 & (0x1U << 20)) ? ~(val) : (val))
#define K21XOR(val) ((keyFrom00To27 & (0x1U << 21)) ? ~(val) : (val))
#define K22XOR(val) ((keyFrom00To27 & (0x1U << 22)) ? ~(val) : (val))
#define K23XOR(val) ((keyFrom00To27 & (0x1U << 23)) ? ~(val) : (val))
#define K24XOR(val) ((keyFrom00To27 & (0x1U << 24)) ? ~(val) : (val))
#define K25XOR(val) ((keyFrom00To27 & (0x1U << 25)) ? ~(val) : (val))
#define K26XOR(val) ((keyFrom00To27 & (0x1U << 26)) ? ~(val) : (val))
#define K27XOR(val) ((keyFrom00To27 & (0x1U << 27)) ? ~(val) : (val))
#define K28XOR(val) ((keyFrom28To48 & (0x1U << (28 - 28))) ? ~(val) : (val))
#define K29XOR(val) ((keyFrom28To48 & (0x1U << (29 - 28))) ? ~(val) : (val))
#define K30XOR(val) ((keyFrom28To48 & (0x1U << (30 - 28))) ? ~(val) : (val))
#define K31XOR(val) ((keyFrom28To48 & (0x1U << (31 - 28))) ? ~(val) : (val))
#define K32XOR(val) ((keyFrom28To48 & (0x1U << (32 - 28))) ? ~(val) : (val))
#define K33XOR(val) ((keyFrom28To48 & (0x1U << (33 - 28))) ? ~(val) : (val))
#define K34XOR(val) ((keyFrom28To48 & (0x1U << (34 - 28))) ? ~(val) : (val))
#define K35XOR(val) ((keyFrom28To48 & (0x1U << (35 - 28))) ? ~(val) : (val))
#define K36XOR(val) ((keyFrom28To48 & (0x1U << (36 - 28))) ? ~(val) : (val))
#define K37XOR(val) ((keyFrom28To48 & (0x1U << (37 - 28))) ? ~(val) : (val))
#define K38XOR(val) ((keyFrom28To48 & (0x1U << (38 - 28))) ? ~(val) : (val))
#define K39XOR(val) ((keyFrom28To48 & (0x1U << (39 - 28))) ? ~(val) : (val))
#define K40XOR(val) ((keyFrom28To48 & (0x1U << (40 - 28))) ? ~(val) : (val))
#define K41XOR(val) ((keyFrom28To48 & (0x1U << (41 - 28))) ? ~(val) : (val))
#define K42XOR(val) ((keyFrom28To48 & (0x1U << (42 - 28))) ? ~(val) : (val))
#define K43XOR(val) ((keyFrom28To48 & (0x1U << (43 - 28))) ? ~(val) : (val))
#define K44XOR(val) ((keyFrom28To48 & (0x1U << (44 - 28))) ? ~(val) : (val))
#define K45XOR(val) ((keyFrom28To48 & (0x1U << (45 - 28))) ? ~(val) : (val))
#define K46XOR(val) ((keyFrom28To48 & (0x1U << (46 - 28))) ? ~(val) : (val))
#define K47XOR(val) ((keyFrom28To48 & (0x1U << (47 - 28))) ? ~(val) : (val))
#define K48XOR(val) ((keyFrom28To48 & (0x1U << (48 - 28))) ? ~(val) : (val))
#define K49XOR(val) ((val) ^ CUDA_keyFrom49To55Array[0])
#define K50XOR(val) ((val) ^ CUDA_keyFrom49To55Array[1])
#define K51XOR(val) ((val) ^ CUDA_keyFrom49To55Array[2])
#define K52XOR(val) ((val) ^ CUDA_keyFrom49To55Array[3])
#define K53XOR(val) ((val) ^ CUDA_keyFrom49To55Array[4])
#define K54XOR(val) ((val) ^ CUDA_keyFrom49To55Array[5])
#define K55XOR(val) ((val) ^ CUDA_keyFrom49To55Array[6])

#define SALT 0 // intSalt

	volatile DES_Vector DB00 = 0;
	volatile DES_Vector DB01 = 0;
	volatile DES_Vector DB02 = 0;
	volatile DES_Vector DB03 = 0;
	volatile DES_Vector DB04 = 0;
	volatile DES_Vector DB05 = 0;
	volatile DES_Vector DB06 = 0;
	volatile DES_Vector DB07 = 0;
	volatile DES_Vector DB08 = 0;
	volatile DES_Vector DB09 = 0;
	volatile DES_Vector DB10 = 0;
	volatile DES_Vector DB11 = 0;
	volatile DES_Vector DB12 = 0;
	volatile DES_Vector DB13 = 0;
	volatile DES_Vector DB14 = 0;
	volatile DES_Vector DB15 = 0;
	// volatile DES_Vector DB16 = 0;
	// volatile DES_Vector DB17 = 0;
	// volatile DES_Vector DB18 = 0;
	// volatile DES_Vector DB19 = 0;
	// volatile DES_Vector DB20 = 0;
	// volatile DES_Vector DB21 = 0;
	// volatile DES_Vector DB22 = 0;
	// volatile DES_Vector DB23 = 0;
	// volatile DES_Vector DB24 = 0;
	// volatile DES_Vector DB25 = 0;
	// volatile DES_Vector DB26 = 0;
	// volatile DES_Vector DB27 = 0;
	// volatile DES_Vector DB28 = 0;
	// volatile DES_Vector DB29 = 0;
	// volatile DES_Vector DB30 = 0;
	// volatile DES_Vector DB31 = 0;
	// volatile DES_Vector DB32 = 0;
	// volatile DES_Vector DB33 = 0;
	// volatile DES_Vector DB34 = 0;
	// volatile DES_Vector DB35 = 0;
	// volatile DES_Vector DB36 = 0;
	// volatile DES_Vector DB37 = 0;
	// volatile DES_Vector DB38 = 0;
	// volatile DES_Vector DB39 = 0;
	// volatile DES_Vector DB40 = 0;
	// volatile DES_Vector DB41 = 0;
	// volatile DES_Vector DB42 = 0;
	// volatile DES_Vector DB43 = 0;
	// volatile DES_Vector DB44 = 0;
	// volatile DES_Vector DB45 = 0;
	// volatile DES_Vector DB46 = 0;
	// volatile DES_Vector DB47 = 0;
	volatile DES_Vector DB48 = 0;
	volatile DES_Vector DB49 = 0;
	volatile DES_Vector DB50 = 0;
	volatile DES_Vector DB51 = 0;
	volatile DES_Vector DB52 = 0;
	volatile DES_Vector DB53 = 0;
	volatile DES_Vector DB54 = 0;
	volatile DES_Vector DB55 = 0;
	volatile DES_Vector DB56 = 0;
	volatile DES_Vector DB57 = 0;
	volatile DES_Vector DB58 = 0;
	volatile DES_Vector DB59 = 0;
	volatile DES_Vector DB60 = 0;
	volatile DES_Vector DB61 = 0;
	volatile DES_Vector DB62 = 0;
	volatile DES_Vector DB63 = 0;
	// #define DB00 db[00<<DB_SHIFT]
	// #define DB01 db[01<<DB_SHIFT]
	// #define DB02 db[02<<DB_SHIFT]
	// #define DB03 db[03<<DB_SHIFT]
	// #define DB04 db[04<<DB_SHIFT]
	// #define DB05 db[05<<DB_SHIFT]
	// #define DB06 db[06<<DB_SHIFT]
	// #define DB07 db[07<<DB_SHIFT]
	// #define DB08 db[ 8<<DB_SHIFT]
	// #define DB09 db[ 9<<DB_SHIFT]
	// #define DB10 db[10<<DB_SHIFT]
	// #define DB11 db[11<<DB_SHIFT]
	// #define DB12 db[12<<DB_SHIFT]
	// #define DB13 db[13<<DB_SHIFT]
	// #define DB14 db[14<<DB_SHIFT]
	// #define DB15 db[15<<DB_SHIFT]
	#define DB16 db[16<<DB_SHIFT]
	#define DB17 db[17<<DB_SHIFT]
	#define DB18 db[18<<DB_SHIFT]
	#define DB19 db[19<<DB_SHIFT]
	#define DB20 db[20<<DB_SHIFT]
	#define DB21 db[21<<DB_SHIFT]
	#define DB22 db[22<<DB_SHIFT]
	#define DB23 db[23<<DB_SHIFT]
	#define DB24 db[24<<DB_SHIFT]
	#define DB25 db[25<<DB_SHIFT]
	#define DB26 db[26<<DB_SHIFT]
	#define DB27 db[27<<DB_SHIFT]
	#define DB28 db[28<<DB_SHIFT]
	#define DB29 db[29<<DB_SHIFT]
	#define DB30 db[30<<DB_SHIFT]
	#define DB31 db[31<<DB_SHIFT]
	#define DB32 db[32<<DB_SHIFT]
	#define DB33 db[33<<DB_SHIFT]
	#define DB34 db[34<<DB_SHIFT]
	#define DB35 db[35<<DB_SHIFT]
	#define DB36 db[36<<DB_SHIFT]
	#define DB37 db[37<<DB_SHIFT]
	#define DB38 db[38<<DB_SHIFT]
	#define DB39 db[39<<DB_SHIFT]
	#define DB40 db[40<<DB_SHIFT]
	#define DB41 db[41<<DB_SHIFT]
	#define DB42 db[42<<DB_SHIFT]
	#define DB43 db[43<<DB_SHIFT]
	#define DB44 db[44<<DB_SHIFT]
	#define DB45 db[45<<DB_SHIFT]
	#define DB46 db[46<<DB_SHIFT]
	#define DB47 db[47<<DB_SHIFT]
	// #define DB48 db[48<<DB_SHIFT]
	// #define DB49 db[49<<DB_SHIFT]
	// #define DB50 db[50<<DB_SHIFT]
	// #define DB51 db[51<<DB_SHIFT]
	// #define DB52 db[52<<DB_SHIFT]
	// #define DB53 db[53<<DB_SHIFT]
	// #define DB54 db[54<<DB_SHIFT]
	// #define DB55 db[55<<DB_SHIFT]
	// #define DB56 db[56<<DB_SHIFT]
	// #define DB57 db[57<<DB_SHIFT]
	// #define DB58 db[58<<DB_SHIFT]
	// #define DB59 db[59<<DB_SHIFT]
	// #define DB60 db[60<<DB_SHIFT]
	// #define DB61 db[61<<DB_SHIFT]
	// #define DB62 db[62<<DB_SHIFT]
	// #define DB63 db[63<<DB_SHIFT]

	for (int i = 0; i < 13; ++i) {
		// ROUND_A(0);
		if (!threadIdx.y) {
		s1(K12XOR((   1 & SALT) ? DB15 : DB31), K46XOR((   2 & SALT) ? DB16 : DB00), K33XOR((   4 & SALT) ? DB17 : DB01), K52XOR((   8 & SALT) ? DB18 : DB02), K48XOR((  16 & SALT) ? DB19 : DB03), K20XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K34XOR((  64 & SALT) ? DB19 : DB03), K55XOR(( 128 & SALT) ? DB20 : DB04), K05XOR(( 256 & SALT) ? DB21 : DB05), K13XOR(( 512 & SALT) ? DB22 : DB06), K18XOR((1024 & SALT) ? DB23 : DB07), K40XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K04XOR(                DB07       ), K32XOR(                DB08       ), K26XOR(                DB09       ), K27XOR(                DB10       ), K38XOR(                DB11       ), K54XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K53XOR(                DB11       ), K06XOR(                DB12       ), K31XOR(                DB13       ), K25XOR(                DB14       ), K19XOR(                DB15       ), K41XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K15XOR((   1 & SALT) ? DB31 : DB15), K24XOR((   2 & SALT) ? DB00 : DB16), K28XOR((   4 & SALT) ? DB01 : DB17), K43XOR((   8 & SALT) ? DB02 : DB18), K30XOR((  16 & SALT) ? DB03 : DB19), K03XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K35XOR((  64 & SALT) ? DB03 : DB19), K22XOR(( 128 & SALT) ? DB04 : DB20), K02XOR(( 256 & SALT) ? DB05 : DB21), K44XOR(( 512 & SALT) ? DB06 : DB22), K14XOR((1024 & SALT) ? DB07 : DB23), K23XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K51XOR(                DB23       ), K16XOR(                DB24       ), K29XOR(                DB25       ), K49XOR(                DB26       ), K07XOR(                DB27       ), K17XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K37XOR(                DB27       ), K08XOR(                DB28       ), K09XOR(                DB29       ), K50XOR(                DB30       ), K42XOR(                DB31       ), K21XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(0);
		if (!threadIdx.y) {
		s1(K05XOR((   1 & SALT) ? DB47 : DB63), K39XOR((   2 & SALT) ? DB48 : DB32), K26XOR((   4 & SALT) ? DB49 : DB33), K45XOR((   8 & SALT) ? DB50 : DB34), K41XOR((  16 & SALT) ? DB51 : DB35), K13XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K27XOR((  64 & SALT) ? DB51 : DB35), K48XOR(( 128 & SALT) ? DB52 : DB36), K53XOR(( 256 & SALT) ? DB53 : DB37), K06XOR(( 512 & SALT) ? DB54 : DB38), K11XOR((1024 & SALT) ? DB55 : DB39), K33XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K52XOR(DB39), K25XOR(DB40), K19XOR(DB41), K20XOR(DB42), K31XOR(DB43), K47XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K46XOR(DB43), K54XOR(DB44), K55XOR(DB45), K18XOR(DB46), K12XOR(DB47), K34XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K08XOR((   1 & SALT) ? DB63 : DB47), K17XOR((   2 & SALT) ? DB32 : DB48), K21XOR((   4 & SALT) ? DB33 : DB49), K36XOR((   8 & SALT) ? DB34 : DB50), K23XOR((  16 & SALT) ? DB35 : DB51), K49XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K28XOR((  64 & SALT) ? DB35 : DB51), K15XOR(( 128 & SALT) ? DB36 : DB52), K24XOR(( 256 & SALT) ? DB37 : DB53), K37XOR(( 512 & SALT) ? DB38 : DB54), K07XOR((1024 & SALT) ? DB39 : DB55), K16XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K44XOR(DB55), K09XOR(DB56), K22XOR(DB57), K42XOR(DB58), K00XOR(DB59), K10XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K30XOR(DB59), K01XOR(DB60), K02XOR(DB61), K43XOR(DB62), K35XOR(DB63), K14XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(96);
		if (!threadIdx.y) {
		s1(K46XOR((   1 & SALT) ? DB15 : DB31), K25XOR((   2 & SALT) ? DB16 : DB00), K12XOR((   4 & SALT) ? DB17 : DB01), K31XOR((   8 & SALT) ? DB18 : DB02), K27XOR((  16 & SALT) ? DB19 : DB03), K54XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K13XOR((  64 & SALT) ? DB19 : DB03), K34XOR(( 128 & SALT) ? DB20 : DB04), K39XOR(( 256 & SALT) ? DB21 : DB05), K47XOR(( 512 & SALT) ? DB22 : DB06), K52XOR((1024 & SALT) ? DB23 : DB07), K19XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K38XOR(                DB07       ), K11XOR(                DB08       ), K05XOR(                DB09       ), K06XOR(                DB10       ), K48XOR(                DB11       ), K33XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K32XOR(                DB11       ), K40XOR(                DB12       ), K41XOR(                DB13       ), K04XOR(                DB14       ), K53XOR(                DB15       ), K20XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K51XOR((   1 & SALT) ? DB31 : DB15), K03XOR((   2 & SALT) ? DB00 : DB16), K07XOR((   4 & SALT) ? DB01 : DB17), K22XOR((   8 & SALT) ? DB02 : DB18), K09XOR((  16 & SALT) ? DB03 : DB19), K35XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K14XOR((  64 & SALT) ? DB03 : DB19), K01XOR(( 128 & SALT) ? DB04 : DB20), K10XOR(( 256 & SALT) ? DB05 : DB21), K23XOR(( 512 & SALT) ? DB06 : DB22), K50XOR((1024 & SALT) ? DB07 : DB23), K02XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K30XOR(                DB23       ), K24XOR(                DB24       ), K08XOR(                DB25       ), K28XOR(                DB26       ), K43XOR(                DB27       ), K49XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K16XOR(                DB27       ), K44XOR(                DB28       ), K17XOR(                DB29       ), K29XOR(                DB30       ), K21XOR(                DB31       ), K00XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(96);
		if (!threadIdx.y) {
		s1(K32XOR((   1 & SALT) ? DB47 : DB63), K11XOR((   2 & SALT) ? DB48 : DB32), K53XOR((   4 & SALT) ? DB49 : DB33), K48XOR((   8 & SALT) ? DB50 : DB34), K13XOR((  16 & SALT) ? DB51 : DB35), K40XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K54XOR((  64 & SALT) ? DB51 : DB35), K20XOR(( 128 & SALT) ? DB52 : DB36), K25XOR(( 256 & SALT) ? DB53 : DB37), K33XOR(( 512 & SALT) ? DB54 : DB38), K38XOR((1024 & SALT) ? DB55 : DB39), K05XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K55XOR(DB39), K52XOR(DB40), K46XOR(DB41), K47XOR(DB42), K34XOR(DB43), K19XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K18XOR(DB43), K26XOR(DB44), K27XOR(DB45), K45XOR(DB46), K39XOR(DB47), K06XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K37XOR((   1 & SALT) ? DB63 : DB47), K42XOR((   2 & SALT) ? DB32 : DB48), K50XOR((   4 & SALT) ? DB33 : DB49), K08XOR((   8 & SALT) ? DB34 : DB50), K24XOR((  16 & SALT) ? DB35 : DB51), K21XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K00XOR((  64 & SALT) ? DB35 : DB51), K44XOR(( 128 & SALT) ? DB36 : DB52), K49XOR(( 256 & SALT) ? DB37 : DB53), K09XOR(( 512 & SALT) ? DB38 : DB54), K36XOR((1024 & SALT) ? DB39 : DB55), K17XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K16XOR(DB55), K10XOR(DB56), K51XOR(DB57), K14XOR(DB58), K29XOR(DB59), K35XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K02XOR(DB59), K30XOR(DB60), K03XOR(DB61), K15XOR(DB62), K07XOR(DB63), K43XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(192);
		if (!threadIdx.y) {
		s1(K18XOR((   1 & SALT) ? DB15 : DB31), K52XOR((   2 & SALT) ? DB16 : DB00), K39XOR((   4 & SALT) ? DB17 : DB01), K34XOR((   8 & SALT) ? DB18 : DB02), K54XOR((  16 & SALT) ? DB19 : DB03), K26XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K40XOR((  64 & SALT) ? DB19 : DB03), K06XOR(( 128 & SALT) ? DB20 : DB04), K11XOR(( 256 & SALT) ? DB21 : DB05), K19XOR(( 512 & SALT) ? DB22 : DB06), K55XOR((1024 & SALT) ? DB23 : DB07), K46XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K41XOR(                DB07       ), K38XOR(                DB08       ), K32XOR(                DB09       ), K33XOR(                DB10       ), K20XOR(                DB11       ), K05XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K04XOR(                DB11       ), K12XOR(                DB12       ), K13XOR(                DB13       ), K31XOR(                DB14       ), K25XOR(                DB15       ), K47XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K23XOR((   1 & SALT) ? DB31 : DB15), K28XOR((   2 & SALT) ? DB00 : DB16), K36XOR((   4 & SALT) ? DB01 : DB17), K51XOR((   8 & SALT) ? DB02 : DB18), K10XOR((  16 & SALT) ? DB03 : DB19), K07XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K43XOR((  64 & SALT) ? DB03 : DB19), K30XOR(( 128 & SALT) ? DB04 : DB20), K35XOR(( 256 & SALT) ? DB05 : DB21), K24XOR(( 512 & SALT) ? DB06 : DB22), K22XOR((1024 & SALT) ? DB07 : DB23), K03XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K02XOR(                DB23       ), K49XOR(                DB24       ), K37XOR(                DB25       ), K00XOR(                DB26       ), K15XOR(                DB27       ), K21XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K17XOR(                DB27       ), K16XOR(                DB28       ), K42XOR(                DB29       ), K01XOR(                DB30       ), K50XOR(                DB31       ), K29XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(192);
		if (!threadIdx.y) {
		s1(K04XOR((   1 & SALT) ? DB47 : DB63), K38XOR((   2 & SALT) ? DB48 : DB32), K25XOR((   4 & SALT) ? DB49 : DB33), K20XOR((   8 & SALT) ? DB50 : DB34), K40XOR((  16 & SALT) ? DB51 : DB35), K12XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K26XOR((  64 & SALT) ? DB51 : DB35), K47XOR(( 128 & SALT) ? DB52 : DB36), K52XOR(( 256 & SALT) ? DB53 : DB37), K05XOR(( 512 & SALT) ? DB54 : DB38), K41XOR((1024 & SALT) ? DB55 : DB39), K32XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K27XOR(DB39), K55XOR(DB40), K18XOR(DB41), K19XOR(DB42), K06XOR(DB43), K46XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K45XOR(DB43), K53XOR(DB44), K54XOR(DB45), K48XOR(DB46), K11XOR(DB47), K33XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K09XOR((   1 & SALT) ? DB63 : DB47), K14XOR((   2 & SALT) ? DB32 : DB48), K22XOR((   4 & SALT) ? DB33 : DB49), K37XOR((   8 & SALT) ? DB34 : DB50), K49XOR((  16 & SALT) ? DB35 : DB51), K50XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K29XOR((  64 & SALT) ? DB35 : DB51), K16XOR(( 128 & SALT) ? DB36 : DB52), K21XOR(( 256 & SALT) ? DB37 : DB53), K10XOR(( 512 & SALT) ? DB38 : DB54), K08XOR((1024 & SALT) ? DB39 : DB55), K42XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K17XOR(DB55), K35XOR(DB56), K23XOR(DB57), K43XOR(DB58), K01XOR(DB59), K07XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K03XOR(DB59), K02XOR(DB60), K28XOR(DB61), K44XOR(DB62), K36XOR(DB63), K15XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(288);
		if (!threadIdx.y) {
		s1(K45XOR((   1 & SALT) ? DB15 : DB31), K55XOR((   2 & SALT) ? DB16 : DB00), K11XOR((   4 & SALT) ? DB17 : DB01), K06XOR((   8 & SALT) ? DB18 : DB02), K26XOR((  16 & SALT) ? DB19 : DB03), K53XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K12XOR((  64 & SALT) ? DB19 : DB03), K33XOR(( 128 & SALT) ? DB20 : DB04), K38XOR(( 256 & SALT) ? DB21 : DB05), K46XOR(( 512 & SALT) ? DB22 : DB06), K27XOR((1024 & SALT) ? DB23 : DB07), K18XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K13XOR(                DB07       ), K41XOR(                DB08       ), K04XOR(                DB09       ), K05XOR(                DB10       ), K47XOR(                DB11       ), K32XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K31XOR(                DB11       ), K39XOR(                DB12       ), K40XOR(                DB13       ), K34XOR(                DB14       ), K52XOR(                DB15       ), K19XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K24XOR((   1 & SALT) ? DB31 : DB15), K00XOR((   2 & SALT) ? DB00 : DB16), K08XOR((   4 & SALT) ? DB01 : DB17), K23XOR((   8 & SALT) ? DB02 : DB18), K35XOR((  16 & SALT) ? DB03 : DB19), K36XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K15XOR((  64 & SALT) ? DB03 : DB19), K02XOR(( 128 & SALT) ? DB04 : DB20), K07XOR(( 256 & SALT) ? DB05 : DB21), K49XOR(( 512 & SALT) ? DB06 : DB22), K51XOR((1024 & SALT) ? DB07 : DB23), K28XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K03XOR(                DB23       ), K21XOR(                DB24       ), K09XOR(                DB25       ), K29XOR(                DB26       ), K44XOR(                DB27       ), K50XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K42XOR(                DB27       ), K17XOR(                DB28       ), K14XOR(                DB29       ), K30XOR(                DB30       ), K22XOR(                DB31       ), K01XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(288);
		if (!threadIdx.y) {
		s1(K31XOR((   1 & SALT) ? DB47 : DB63), K41XOR((   2 & SALT) ? DB48 : DB32), K52XOR((   4 & SALT) ? DB49 : DB33), K47XOR((   8 & SALT) ? DB50 : DB34), K12XOR((  16 & SALT) ? DB51 : DB35), K39XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K53XOR((  64 & SALT) ? DB51 : DB35), K19XOR(( 128 & SALT) ? DB52 : DB36), K55XOR(( 256 & SALT) ? DB53 : DB37), K32XOR(( 512 & SALT) ? DB54 : DB38), K13XOR((1024 & SALT) ? DB55 : DB39), K04XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K54XOR(DB39), K27XOR(DB40), K45XOR(DB41), K46XOR(DB42), K33XOR(DB43), K18XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K48XOR(DB43), K25XOR(DB44), K26XOR(DB45), K20XOR(DB46), K38XOR(DB47), K05XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K10XOR((   1 & SALT) ? DB63 : DB47), K43XOR((   2 & SALT) ? DB32 : DB48), K51XOR((   4 & SALT) ? DB33 : DB49), K09XOR((   8 & SALT) ? DB34 : DB50), K21XOR((  16 & SALT) ? DB35 : DB51), K22XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K01XOR((  64 & SALT) ? DB35 : DB51), K17XOR(( 128 & SALT) ? DB36 : DB52), K50XOR(( 256 & SALT) ? DB37 : DB53), K35XOR(( 512 & SALT) ? DB38 : DB54), K37XOR((1024 & SALT) ? DB39 : DB55), K14XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K42XOR(DB55), K07XOR(DB56), K24XOR(DB57), K15XOR(DB58), K30XOR(DB59), K36XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K28XOR(DB59), K03XOR(DB60), K00XOR(DB61), K16XOR(DB62), K08XOR(DB63), K44XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(384);
		if (!threadIdx.y) {
		s1(K55XOR((   1 & SALT) ? DB15 : DB31), K34XOR((   2 & SALT) ? DB16 : DB00), K45XOR((   4 & SALT) ? DB17 : DB01), K40XOR((   8 & SALT) ? DB18 : DB02), K05XOR((  16 & SALT) ? DB19 : DB03), K32XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K46XOR((  64 & SALT) ? DB19 : DB03), K12XOR(( 128 & SALT) ? DB20 : DB04), K48XOR(( 256 & SALT) ? DB21 : DB05), K25XOR(( 512 & SALT) ? DB22 : DB06), K06XOR((1024 & SALT) ? DB23 : DB07), K52XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K47XOR(                DB07       ), K20XOR(                DB08       ), K38XOR(                DB09       ), K39XOR(                DB10       ), K26XOR(                DB11       ), K11XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K41XOR(                DB11       ), K18XOR(                DB12       ), K19XOR(                DB13       ), K13XOR(                DB14       ), K31XOR(                DB15       ), K53XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K03XOR((   1 & SALT) ? DB31 : DB15), K36XOR((   2 & SALT) ? DB00 : DB16), K44XOR((   4 & SALT) ? DB01 : DB17), K02XOR((   8 & SALT) ? DB02 : DB18), K14XOR((  16 & SALT) ? DB03 : DB19), K15XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K51XOR((  64 & SALT) ? DB03 : DB19), K10XOR(( 128 & SALT) ? DB04 : DB20), K43XOR(( 256 & SALT) ? DB05 : DB21), K28XOR(( 512 & SALT) ? DB06 : DB22), K30XOR((1024 & SALT) ? DB07 : DB23), K07XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K35XOR(                DB23       ), K00XOR(                DB24       ), K17XOR(                DB25       ), K08XOR(                DB26       ), K23XOR(                DB27       ), K29XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K21XOR(                DB27       ), K49XOR(                DB28       ), K50XOR(                DB29       ), K09XOR(                DB30       ), K01XOR(                DB31       ), K37XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(384);
		if (!threadIdx.y) {
		s1(K41XOR((   1 & SALT) ? DB47 : DB63), K20XOR((   2 & SALT) ? DB48 : DB32), K31XOR((   4 & SALT) ? DB49 : DB33), K26XOR((   8 & SALT) ? DB50 : DB34), K46XOR((  16 & SALT) ? DB51 : DB35), K18XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K32XOR((  64 & SALT) ? DB51 : DB35), K53XOR(( 128 & SALT) ? DB52 : DB36), K34XOR(( 256 & SALT) ? DB53 : DB37), K11XOR(( 512 & SALT) ? DB54 : DB38), K47XOR((1024 & SALT) ? DB55 : DB39), K38XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K33XOR(DB39), K06XOR(DB40), K55XOR(DB41), K25XOR(DB42), K12XOR(DB43), K52XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K27XOR(DB43), K04XOR(DB44), K05XOR(DB45), K54XOR(DB46), K48XOR(DB47), K39XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K42XOR((   1 & SALT) ? DB63 : DB47), K22XOR((   2 & SALT) ? DB32 : DB48), K30XOR((   4 & SALT) ? DB33 : DB49), K17XOR((   8 & SALT) ? DB34 : DB50), K00XOR((  16 & SALT) ? DB35 : DB51), K01XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K37XOR((  64 & SALT) ? DB35 : DB51), K49XOR(( 128 & SALT) ? DB36 : DB52), K29XOR(( 256 & SALT) ? DB37 : DB53), K14XOR(( 512 & SALT) ? DB38 : DB54), K16XOR((1024 & SALT) ? DB39 : DB55), K50XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K21XOR(DB55), K43XOR(DB56), K03XOR(DB57), K51XOR(DB58), K09XOR(DB59), K15XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K07XOR(DB59), K35XOR(DB60), K36XOR(DB61), K24XOR(DB62), K44XOR(DB63), K23XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(480);
		if (!threadIdx.y) {
		s1(K27XOR((   1 & SALT) ? DB15 : DB31), K06XOR((   2 & SALT) ? DB16 : DB00), K48XOR((   4 & SALT) ? DB17 : DB01), K12XOR((   8 & SALT) ? DB18 : DB02), K32XOR((  16 & SALT) ? DB19 : DB03), K04XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K18XOR((  64 & SALT) ? DB19 : DB03), K39XOR(( 128 & SALT) ? DB20 : DB04), K20XOR(( 256 & SALT) ? DB21 : DB05), K52XOR(( 512 & SALT) ? DB22 : DB06), K33XOR((1024 & SALT) ? DB23 : DB07), K55XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K19XOR(                DB07       ), K47XOR(                DB08       ), K41XOR(                DB09       ), K11XOR(                DB10       ), K53XOR(                DB11       ), K38XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K13XOR(                DB11       ), K45XOR(                DB12       ), K46XOR(                DB13       ), K40XOR(                DB14       ), K34XOR(                DB15       ), K25XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K28XOR((   1 & SALT) ? DB31 : DB15), K08XOR((   2 & SALT) ? DB00 : DB16), K16XOR((   4 & SALT) ? DB01 : DB17), K03XOR((   8 & SALT) ? DB02 : DB18), K43XOR((  16 & SALT) ? DB03 : DB19), K44XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K23XOR((  64 & SALT) ? DB03 : DB19), K35XOR(( 128 & SALT) ? DB04 : DB20), K15XOR(( 256 & SALT) ? DB05 : DB21), K00XOR(( 512 & SALT) ? DB06 : DB22), K02XOR((1024 & SALT) ? DB07 : DB23), K36XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K07XOR(                DB23       ), K29XOR(                DB24       ), K42XOR(                DB25       ), K37XOR(                DB26       ), K24XOR(                DB27       ), K01XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K50XOR(                DB27       ), K21XOR(                DB28       ), K22XOR(                DB29       ), K10XOR(                DB30       ), K30XOR(                DB31       ), K09XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(480);
		if (!threadIdx.y) {
		s1(K13XOR((   1 & SALT) ? DB47 : DB63), K47XOR((   2 & SALT) ? DB48 : DB32), K34XOR((   4 & SALT) ? DB49 : DB33), K53XOR((   8 & SALT) ? DB50 : DB34), K18XOR((  16 & SALT) ? DB51 : DB35), K45XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K04XOR((  64 & SALT) ? DB51 : DB35), K25XOR(( 128 & SALT) ? DB52 : DB36), K06XOR(( 256 & SALT) ? DB53 : DB37), K38XOR(( 512 & SALT) ? DB54 : DB38), K19XOR((1024 & SALT) ? DB55 : DB39), K41XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K05XOR(DB39), K33XOR(DB40), K27XOR(DB41), K52XOR(DB42), K39XOR(DB43), K55XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K54XOR(DB43), K31XOR(DB44), K32XOR(DB45), K26XOR(DB46), K20XOR(DB47), K11XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K14XOR((   1 & SALT) ? DB63 : DB47), K51XOR((   2 & SALT) ? DB32 : DB48), K02XOR((   4 & SALT) ? DB33 : DB49), K42XOR((   8 & SALT) ? DB34 : DB50), K29XOR((  16 & SALT) ? DB35 : DB51), K30XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K09XOR((  64 & SALT) ? DB35 : DB51), K21XOR(( 128 & SALT) ? DB36 : DB52), K01XOR(( 256 & SALT) ? DB37 : DB53), K43XOR(( 512 & SALT) ? DB38 : DB54), K17XOR((1024 & SALT) ? DB39 : DB55), K22XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K50XOR(DB55), K15XOR(DB56), K28XOR(DB57), K23XOR(DB58), K10XOR(DB59), K44XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K36XOR(DB59), K07XOR(DB60), K08XOR(DB61), K49XOR(DB62), K16XOR(DB63), K24XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(576);
		if (!threadIdx.y) {
		s1(K54XOR((   1 & SALT) ? DB15 : DB31), K33XOR((   2 & SALT) ? DB16 : DB00), K20XOR((   4 & SALT) ? DB17 : DB01), K39XOR((   8 & SALT) ? DB18 : DB02), K04XOR((  16 & SALT) ? DB19 : DB03), K31XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K45XOR((  64 & SALT) ? DB19 : DB03), K11XOR(( 128 & SALT) ? DB20 : DB04), K47XOR(( 256 & SALT) ? DB21 : DB05), K55XOR(( 512 & SALT) ? DB22 : DB06), K05XOR((1024 & SALT) ? DB23 : DB07), K27XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K46XOR(                DB07       ), K19XOR(                DB08       ), K13XOR(                DB09       ), K38XOR(                DB10       ), K25XOR(                DB11       ), K41XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K40XOR(                DB11       ), K48XOR(                DB12       ), K18XOR(                DB13       ), K12XOR(                DB14       ), K06XOR(                DB15       ), K52XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K00XOR((   1 & SALT) ? DB31 : DB15), K37XOR((   2 & SALT) ? DB00 : DB16), K17XOR((   4 & SALT) ? DB01 : DB17), K28XOR((   8 & SALT) ? DB02 : DB18), K15XOR((  16 & SALT) ? DB03 : DB19), K16XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K24XOR((  64 & SALT) ? DB03 : DB19), K07XOR(( 128 & SALT) ? DB04 : DB20), K44XOR(( 256 & SALT) ? DB05 : DB21), K29XOR(( 512 & SALT) ? DB06 : DB22), K03XOR((1024 & SALT) ? DB07 : DB23), K08XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K36XOR(                DB23       ), K01XOR(                DB24       ), K14XOR(                DB25       ), K09XOR(                DB26       ), K49XOR(                DB27       ), K30XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K22XOR(                DB27       ), K50XOR(                DB28       ), K51XOR(                DB29       ), K35XOR(                DB30       ), K02XOR(                DB31       ), K10XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(576);
		if (!threadIdx.y) {
		s1(K40XOR((   1 & SALT) ? DB47 : DB63), K19XOR((   2 & SALT) ? DB48 : DB32), K06XOR((   4 & SALT) ? DB49 : DB33), K25XOR((   8 & SALT) ? DB50 : DB34), K45XOR((  16 & SALT) ? DB51 : DB35), K48XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K31XOR((  64 & SALT) ? DB51 : DB35), K52XOR(( 128 & SALT) ? DB52 : DB36), K33XOR(( 256 & SALT) ? DB53 : DB37), K41XOR(( 512 & SALT) ? DB54 : DB38), K46XOR((1024 & SALT) ? DB55 : DB39), K13XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K32XOR(DB39), K05XOR(DB40), K54XOR(DB41), K55XOR(DB42), K11XOR(DB43), K27XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K26XOR(DB43), K34XOR(DB44), K04XOR(DB45), K53XOR(DB46), K47XOR(DB47), K38XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K43XOR((   1 & SALT) ? DB63 : DB47), K23XOR((   2 & SALT) ? DB32 : DB48), K03XOR((   4 & SALT) ? DB33 : DB49), K14XOR((   8 & SALT) ? DB34 : DB50), K01XOR((  16 & SALT) ? DB35 : DB51), K02XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K10XOR((  64 & SALT) ? DB35 : DB51), K50XOR(( 128 & SALT) ? DB36 : DB52), K30XOR(( 256 & SALT) ? DB37 : DB53), K15XOR(( 512 & SALT) ? DB38 : DB54), K42XOR((1024 & SALT) ? DB39 : DB55), K51XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K22XOR(DB55), K44XOR(DB56), K00XOR(DB57), K24XOR(DB58), K35XOR(DB59), K16XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K08XOR(DB59), K36XOR(DB60), K37XOR(DB61), K21XOR(DB62), K17XOR(DB63), K49XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(672);
		if (!threadIdx.y) {
		s1(K26XOR((   1 & SALT) ? DB15 : DB31), K05XOR((   2 & SALT) ? DB16 : DB00), K47XOR((   4 & SALT) ? DB17 : DB01), K11XOR((   8 & SALT) ? DB18 : DB02), K31XOR((  16 & SALT) ? DB19 : DB03), K34XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K48XOR((  64 & SALT) ? DB19 : DB03), K38XOR(( 128 & SALT) ? DB20 : DB04), K19XOR(( 256 & SALT) ? DB21 : DB05), K27XOR(( 512 & SALT) ? DB22 : DB06), K32XOR((1024 & SALT) ? DB23 : DB07), K54XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K18XOR(                DB07       ), K46XOR(                DB08       ), K40XOR(                DB09       ), K41XOR(                DB10       ), K52XOR(                DB11       ), K13XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K12XOR(                DB11       ), K20XOR(                DB12       ), K45XOR(                DB13       ), K39XOR(                DB14       ), K33XOR(                DB15       ), K55XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K29XOR((   1 & SALT) ? DB31 : DB15), K09XOR((   2 & SALT) ? DB00 : DB16), K42XOR((   4 & SALT) ? DB01 : DB17), K00XOR((   8 & SALT) ? DB02 : DB18), K44XOR((  16 & SALT) ? DB03 : DB19), K17XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K49XOR((  64 & SALT) ? DB03 : DB19), K36XOR(( 128 & SALT) ? DB04 : DB20), K16XOR(( 256 & SALT) ? DB05 : DB21), K01XOR(( 512 & SALT) ? DB06 : DB22), K28XOR((1024 & SALT) ? DB07 : DB23), K37XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K08XOR(                DB23       ), K30XOR(                DB24       ), K43XOR(                DB25       ), K10XOR(                DB26       ), K21XOR(                DB27       ), K02XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K51XOR(                DB27       ), K22XOR(                DB28       ), K23XOR(                DB29       ), K07XOR(                DB30       ), K03XOR(                DB31       ), K35XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(672);
		if (!threadIdx.y) {
		s1(K19XOR((   1 & SALT) ? DB47 : DB63), K53XOR((   2 & SALT) ? DB48 : DB32), K40XOR((   4 & SALT) ? DB49 : DB33), K04XOR((   8 & SALT) ? DB50 : DB34), K55XOR((  16 & SALT) ? DB51 : DB35), K27XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K41XOR((  64 & SALT) ? DB51 : DB35), K31XOR(( 128 & SALT) ? DB52 : DB36), K12XOR(( 256 & SALT) ? DB53 : DB37), K20XOR(( 512 & SALT) ? DB54 : DB38), K25XOR((1024 & SALT) ? DB55 : DB39), K47XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K11XOR(DB39), K39XOR(DB40), K33XOR(DB41), K34XOR(DB42), K45XOR(DB43), K06XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K05XOR(DB43), K13XOR(DB44), K38XOR(DB45), K32XOR(DB46), K26XOR(DB47), K48XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K22XOR((   1 & SALT) ? DB63 : DB47), K02XOR((   2 & SALT) ? DB32 : DB48), K35XOR((   4 & SALT) ? DB33 : DB49), K50XOR((   8 & SALT) ? DB34 : DB50), K37XOR((  16 & SALT) ? DB35 : DB51), K10XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K42XOR((  64 & SALT) ? DB35 : DB51), K29XOR(( 128 & SALT) ? DB36 : DB52), K09XOR(( 256 & SALT) ? DB37 : DB53), K51XOR(( 512 & SALT) ? DB38 : DB54), K21XOR((1024 & SALT) ? DB39 : DB55), K30XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K01XOR(DB55), K23XOR(DB56), K36XOR(DB57), K03XOR(DB58), K14XOR(DB59), K24XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K44XOR(DB59), K15XOR(DB60), K16XOR(DB61), K00XOR(DB62), K49XOR(DB63), K28XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		if (i >= 12)
			break;

		// ROUND_B(-48);
		if (!threadIdx.y) {
		s1(K12XOR((   1 & SALT) ? DB47 : DB63), K46XOR((   2 & SALT) ? DB48 : DB32), K33XOR((   4 & SALT) ? DB49 : DB33), K52XOR((   8 & SALT) ? DB50 : DB34), K48XOR((  16 & SALT) ? DB51 : DB35), K20XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K34XOR((  64 & SALT) ? DB51 : DB35), K55XOR(( 128 & SALT) ? DB52 : DB36), K05XOR(( 256 & SALT) ? DB53 : DB37), K13XOR(( 512 & SALT) ? DB54 : DB38), K18XOR((1024 & SALT) ? DB55 : DB39), K40XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K04XOR(DB39), K32XOR(DB40), K26XOR(DB41), K27XOR(DB42), K38XOR(DB43), K54XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K53XOR(DB43), K06XOR(DB44), K31XOR(DB45), K25XOR(DB46), K19XOR(DB47), K41XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K15XOR((   1 & SALT) ? DB63 : DB47), K24XOR((   2 & SALT) ? DB32 : DB48), K28XOR((   4 & SALT) ? DB33 : DB49), K43XOR((   8 & SALT) ? DB34 : DB50), K30XOR((  16 & SALT) ? DB35 : DB51), K03XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K35XOR((  64 & SALT) ? DB35 : DB51), K22XOR(( 128 & SALT) ? DB36 : DB52), K02XOR(( 256 & SALT) ? DB37 : DB53), K44XOR(( 512 & SALT) ? DB38 : DB54), K14XOR((1024 & SALT) ? DB39 : DB55), K23XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K51XOR(DB55), K16XOR(DB56), K29XOR(DB57), K49XOR(DB58), K07XOR(DB59), K17XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K37XOR(DB59), K08XOR(DB60), K09XOR(DB61), K50XOR(DB62), K42XOR(DB63), K21XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(48);
		if (!threadIdx.y) {
		s1(K05XOR((   1 & SALT) ? DB15 : DB31), K39XOR((   2 & SALT) ? DB16 : DB00), K26XOR((   4 & SALT) ? DB17 : DB01), K45XOR((   8 & SALT) ? DB18 : DB02), K41XOR((  16 & SALT) ? DB19 : DB03), K13XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K27XOR((  64 & SALT) ? DB19 : DB03), K48XOR(( 128 & SALT) ? DB20 : DB04), K53XOR(( 256 & SALT) ? DB21 : DB05), K06XOR(( 512 & SALT) ? DB22 : DB06), K11XOR((1024 & SALT) ? DB23 : DB07), K33XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K52XOR(                DB07       ), K25XOR(                DB08       ), K19XOR(                DB09       ), K20XOR(                DB10       ), K31XOR(                DB11       ), K47XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K46XOR(                DB11       ), K54XOR(                DB12       ), K55XOR(                DB13       ), K18XOR(                DB14       ), K12XOR(                DB15       ), K34XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K08XOR((   1 & SALT) ? DB31 : DB15), K17XOR((   2 & SALT) ? DB00 : DB16), K21XOR((   4 & SALT) ? DB01 : DB17), K36XOR((   8 & SALT) ? DB02 : DB18), K23XOR((  16 & SALT) ? DB03 : DB19), K49XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K28XOR((  64 & SALT) ? DB03 : DB19), K15XOR(( 128 & SALT) ? DB04 : DB20), K24XOR(( 256 & SALT) ? DB05 : DB21), K37XOR(( 512 & SALT) ? DB06 : DB22), K07XOR((1024 & SALT) ? DB07 : DB23), K16XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K44XOR(                DB23       ), K09XOR(                DB24       ), K22XOR(                DB25       ), K42XOR(                DB26       ), K00XOR(                DB27       ), K10XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K30XOR(                DB27       ), K01XOR(                DB28       ), K02XOR(                DB29       ), K43XOR(                DB30       ), K35XOR(                DB31       ), K14XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(48);
		if (!threadIdx.y) {
		s1(K46XOR((   1 & SALT) ? DB47 : DB63), K25XOR((   2 & SALT) ? DB48 : DB32), K12XOR((   4 & SALT) ? DB49 : DB33), K31XOR((   8 & SALT) ? DB50 : DB34), K27XOR((  16 & SALT) ? DB51 : DB35), K54XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K13XOR((  64 & SALT) ? DB51 : DB35), K34XOR(( 128 & SALT) ? DB52 : DB36), K39XOR(( 256 & SALT) ? DB53 : DB37), K47XOR(( 512 & SALT) ? DB54 : DB38), K52XOR((1024 & SALT) ? DB55 : DB39), K19XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K38XOR(DB39), K11XOR(DB40), K05XOR(DB41), K06XOR(DB42), K48XOR(DB43), K33XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K32XOR(DB43), K40XOR(DB44), K41XOR(DB45), K04XOR(DB46), K53XOR(DB47), K20XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K51XOR((   1 & SALT) ? DB63 : DB47), K03XOR((   2 & SALT) ? DB32 : DB48), K07XOR((   4 & SALT) ? DB33 : DB49), K22XOR((   8 & SALT) ? DB34 : DB50), K09XOR((  16 & SALT) ? DB35 : DB51), K35XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K14XOR((  64 & SALT) ? DB35 : DB51), K01XOR(( 128 & SALT) ? DB36 : DB52), K10XOR(( 256 & SALT) ? DB37 : DB53), K23XOR(( 512 & SALT) ? DB38 : DB54), K50XOR((1024 & SALT) ? DB39 : DB55), K02XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K30XOR(DB55), K24XOR(DB56), K08XOR(DB57), K28XOR(DB58), K43XOR(DB59), K49XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K16XOR(DB59), K44XOR(DB60), K17XOR(DB61), K29XOR(DB62), K21XOR(DB63), K00XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(144);
		if (!threadIdx.y) {
		s1(K32XOR((   1 & SALT) ? DB15 : DB31), K11XOR((   2 & SALT) ? DB16 : DB00), K53XOR((   4 & SALT) ? DB17 : DB01), K48XOR((   8 & SALT) ? DB18 : DB02), K13XOR((  16 & SALT) ? DB19 : DB03), K40XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K54XOR((  64 & SALT) ? DB19 : DB03), K20XOR(( 128 & SALT) ? DB20 : DB04), K25XOR(( 256 & SALT) ? DB21 : DB05), K33XOR(( 512 & SALT) ? DB22 : DB06), K38XOR((1024 & SALT) ? DB23 : DB07), K05XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K55XOR(                DB07       ), K52XOR(                DB08       ), K46XOR(                DB09       ), K47XOR(                DB10       ), K34XOR(                DB11       ), K19XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K18XOR(                DB11       ), K26XOR(                DB12       ), K27XOR(                DB13       ), K45XOR(                DB14       ), K39XOR(                DB15       ), K06XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K37XOR((   1 & SALT) ? DB31 : DB15), K42XOR((   2 & SALT) ? DB00 : DB16), K50XOR((   4 & SALT) ? DB01 : DB17), K08XOR((   8 & SALT) ? DB02 : DB18), K24XOR((  16 & SALT) ? DB03 : DB19), K21XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K00XOR((  64 & SALT) ? DB03 : DB19), K44XOR(( 128 & SALT) ? DB04 : DB20), K49XOR(( 256 & SALT) ? DB05 : DB21), K09XOR(( 512 & SALT) ? DB06 : DB22), K36XOR((1024 & SALT) ? DB07 : DB23), K17XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K16XOR(                DB23       ), K10XOR(                DB24       ), K51XOR(                DB25       ), K14XOR(                DB26       ), K29XOR(                DB27       ), K35XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K02XOR(                DB27       ), K30XOR(                DB28       ), K03XOR(                DB29       ), K15XOR(                DB30       ), K07XOR(                DB31       ), K43XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(144);
		if (!threadIdx.y) {
		s1(K18XOR((   1 & SALT) ? DB47 : DB63), K52XOR((   2 & SALT) ? DB48 : DB32), K39XOR((   4 & SALT) ? DB49 : DB33), K34XOR((   8 & SALT) ? DB50 : DB34), K54XOR((  16 & SALT) ? DB51 : DB35), K26XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K40XOR((  64 & SALT) ? DB51 : DB35), K06XOR(( 128 & SALT) ? DB52 : DB36), K11XOR(( 256 & SALT) ? DB53 : DB37), K19XOR(( 512 & SALT) ? DB54 : DB38), K55XOR((1024 & SALT) ? DB55 : DB39), K46XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K41XOR(DB39), K38XOR(DB40), K32XOR(DB41), K33XOR(DB42), K20XOR(DB43), K05XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K04XOR(DB43), K12XOR(DB44), K13XOR(DB45), K31XOR(DB46), K25XOR(DB47), K47XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K23XOR((   1 & SALT) ? DB63 : DB47), K28XOR((   2 & SALT) ? DB32 : DB48), K36XOR((   4 & SALT) ? DB33 : DB49), K51XOR((   8 & SALT) ? DB34 : DB50), K10XOR((  16 & SALT) ? DB35 : DB51), K07XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K43XOR((  64 & SALT) ? DB35 : DB51), K30XOR(( 128 & SALT) ? DB36 : DB52), K35XOR(( 256 & SALT) ? DB37 : DB53), K24XOR(( 512 & SALT) ? DB38 : DB54), K22XOR((1024 & SALT) ? DB39 : DB55), K03XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K02XOR(DB55), K49XOR(DB56), K37XOR(DB57), K00XOR(DB58), K15XOR(DB59), K21XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K17XOR(DB59), K16XOR(DB60), K42XOR(DB61), K01XOR(DB62), K50XOR(DB63), K29XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(240);
		if (!threadIdx.y) {
		s1(K04XOR((   1 & SALT) ? DB15 : DB31), K38XOR((   2 & SALT) ? DB16 : DB00), K25XOR((   4 & SALT) ? DB17 : DB01), K20XOR((   8 & SALT) ? DB18 : DB02), K40XOR((  16 & SALT) ? DB19 : DB03), K12XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K26XOR((  64 & SALT) ? DB19 : DB03), K47XOR(( 128 & SALT) ? DB20 : DB04), K52XOR(( 256 & SALT) ? DB21 : DB05), K05XOR(( 512 & SALT) ? DB22 : DB06), K41XOR((1024 & SALT) ? DB23 : DB07), K32XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K27XOR(                DB07       ), K55XOR(                DB08       ), K18XOR(                DB09       ), K19XOR(                DB10       ), K06XOR(                DB11       ), K46XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K45XOR(                DB11       ), K53XOR(                DB12       ), K54XOR(                DB13       ), K48XOR(                DB14       ), K11XOR(                DB15       ), K33XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K09XOR((   1 & SALT) ? DB31 : DB15), K14XOR((   2 & SALT) ? DB00 : DB16), K22XOR((   4 & SALT) ? DB01 : DB17), K37XOR((   8 & SALT) ? DB02 : DB18), K49XOR((  16 & SALT) ? DB03 : DB19), K50XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K29XOR((  64 & SALT) ? DB03 : DB19), K16XOR(( 128 & SALT) ? DB04 : DB20), K21XOR(( 256 & SALT) ? DB05 : DB21), K10XOR(( 512 & SALT) ? DB06 : DB22), K08XOR((1024 & SALT) ? DB07 : DB23), K42XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K17XOR(                DB23       ), K35XOR(                DB24       ), K23XOR(                DB25       ), K43XOR(                DB26       ), K01XOR(                DB27       ), K07XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K03XOR(                DB27       ), K02XOR(                DB28       ), K28XOR(                DB29       ), K44XOR(                DB30       ), K36XOR(                DB31       ), K15XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(240);
		if (!threadIdx.y) {
		s1(K45XOR((   1 & SALT) ? DB47 : DB63), K55XOR((   2 & SALT) ? DB48 : DB32), K11XOR((   4 & SALT) ? DB49 : DB33), K06XOR((   8 & SALT) ? DB50 : DB34), K26XOR((  16 & SALT) ? DB51 : DB35), K53XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K12XOR((  64 & SALT) ? DB51 : DB35), K33XOR(( 128 & SALT) ? DB52 : DB36), K38XOR(( 256 & SALT) ? DB53 : DB37), K46XOR(( 512 & SALT) ? DB54 : DB38), K27XOR((1024 & SALT) ? DB55 : DB39), K18XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K13XOR(DB39), K41XOR(DB40), K04XOR(DB41), K05XOR(DB42), K47XOR(DB43), K32XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K31XOR(DB43), K39XOR(DB44), K40XOR(DB45), K34XOR(DB46), K52XOR(DB47), K19XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K24XOR((   1 & SALT) ? DB63 : DB47), K00XOR((   2 & SALT) ? DB32 : DB48), K08XOR((   4 & SALT) ? DB33 : DB49), K23XOR((   8 & SALT) ? DB34 : DB50), K35XOR((  16 & SALT) ? DB35 : DB51), K36XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K15XOR((  64 & SALT) ? DB35 : DB51), K02XOR(( 128 & SALT) ? DB36 : DB52), K07XOR(( 256 & SALT) ? DB37 : DB53), K49XOR(( 512 & SALT) ? DB38 : DB54), K51XOR((1024 & SALT) ? DB39 : DB55), K28XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K03XOR(DB55), K21XOR(DB56), K09XOR(DB57), K29XOR(DB58), K44XOR(DB59), K50XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K42XOR(DB59), K17XOR(DB60), K14XOR(DB61), K30XOR(DB62), K22XOR(DB63), K01XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(336);
		if (!threadIdx.y) {
		s1(K31XOR((   1 & SALT) ? DB15 : DB31), K41XOR((   2 & SALT) ? DB16 : DB00), K52XOR((   4 & SALT) ? DB17 : DB01), K47XOR((   8 & SALT) ? DB18 : DB02), K12XOR((  16 & SALT) ? DB19 : DB03), K39XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K53XOR((  64 & SALT) ? DB19 : DB03), K19XOR(( 128 & SALT) ? DB20 : DB04), K55XOR(( 256 & SALT) ? DB21 : DB05), K32XOR(( 512 & SALT) ? DB22 : DB06), K13XOR((1024 & SALT) ? DB23 : DB07), K04XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K54XOR(                DB07       ), K27XOR(                DB08       ), K45XOR(                DB09       ), K46XOR(                DB10       ), K33XOR(                DB11       ), K18XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K48XOR(                DB11       ), K25XOR(                DB12       ), K26XOR(                DB13       ), K20XOR(                DB14       ), K38XOR(                DB15       ), K05XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K10XOR((   1 & SALT) ? DB31 : DB15), K43XOR((   2 & SALT) ? DB00 : DB16), K51XOR((   4 & SALT) ? DB01 : DB17), K09XOR((   8 & SALT) ? DB02 : DB18), K21XOR((  16 & SALT) ? DB03 : DB19), K22XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K01XOR((  64 & SALT) ? DB03 : DB19), K17XOR(( 128 & SALT) ? DB04 : DB20), K50XOR(( 256 & SALT) ? DB05 : DB21), K35XOR(( 512 & SALT) ? DB06 : DB22), K37XOR((1024 & SALT) ? DB07 : DB23), K14XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K42XOR(                DB23       ), K07XOR(                DB24       ), K24XOR(                DB25       ), K15XOR(                DB26       ), K30XOR(                DB27       ), K36XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K28XOR(                DB27       ), K03XOR(                DB28       ), K00XOR(                DB29       ), K16XOR(                DB30       ), K08XOR(                DB31       ), K44XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(336);
		if (!threadIdx.y) {
		s1(K55XOR((   1 & SALT) ? DB47 : DB63), K34XOR((   2 & SALT) ? DB48 : DB32), K45XOR((   4 & SALT) ? DB49 : DB33), K40XOR((   8 & SALT) ? DB50 : DB34), K05XOR((  16 & SALT) ? DB51 : DB35), K32XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K46XOR((  64 & SALT) ? DB51 : DB35), K12XOR(( 128 & SALT) ? DB52 : DB36), K48XOR(( 256 & SALT) ? DB53 : DB37), K25XOR(( 512 & SALT) ? DB54 : DB38), K06XOR((1024 & SALT) ? DB55 : DB39), K52XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K47XOR(DB39), K20XOR(DB40), K38XOR(DB41), K39XOR(DB42), K26XOR(DB43), K11XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K41XOR(DB43), K18XOR(DB44), K19XOR(DB45), K13XOR(DB46), K31XOR(DB47), K53XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K03XOR((   1 & SALT) ? DB63 : DB47), K36XOR((   2 & SALT) ? DB32 : DB48), K44XOR((   4 & SALT) ? DB33 : DB49), K02XOR((   8 & SALT) ? DB34 : DB50), K14XOR((  16 & SALT) ? DB35 : DB51), K15XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K51XOR((  64 & SALT) ? DB35 : DB51), K10XOR(( 128 & SALT) ? DB36 : DB52), K43XOR(( 256 & SALT) ? DB37 : DB53), K28XOR(( 512 & SALT) ? DB38 : DB54), K30XOR((1024 & SALT) ? DB39 : DB55), K07XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K35XOR(DB55), K00XOR(DB56), K17XOR(DB57), K08XOR(DB58), K23XOR(DB59), K29XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K21XOR(DB59), K49XOR(DB60), K50XOR(DB61), K09XOR(DB62), K01XOR(DB63), K37XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(432);
		if (!threadIdx.y) {
		s1(K41XOR((   1 & SALT) ? DB15 : DB31), K20XOR((   2 & SALT) ? DB16 : DB00), K31XOR((   4 & SALT) ? DB17 : DB01), K26XOR((   8 & SALT) ? DB18 : DB02), K46XOR((  16 & SALT) ? DB19 : DB03), K18XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K32XOR((  64 & SALT) ? DB19 : DB03), K53XOR(( 128 & SALT) ? DB20 : DB04), K34XOR(( 256 & SALT) ? DB21 : DB05), K11XOR(( 512 & SALT) ? DB22 : DB06), K47XOR((1024 & SALT) ? DB23 : DB07), K38XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K33XOR(                DB07       ), K06XOR(                DB08       ), K55XOR(                DB09       ), K25XOR(                DB10       ), K12XOR(                DB11       ), K52XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K27XOR(                DB11       ), K04XOR(                DB12       ), K05XOR(                DB13       ), K54XOR(                DB14       ), K48XOR(                DB15       ), K39XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K42XOR((   1 & SALT) ? DB31 : DB15), K22XOR((   2 & SALT) ? DB00 : DB16), K30XOR((   4 & SALT) ? DB01 : DB17), K17XOR((   8 & SALT) ? DB02 : DB18), K00XOR((  16 & SALT) ? DB03 : DB19), K01XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K37XOR((  64 & SALT) ? DB03 : DB19), K49XOR(( 128 & SALT) ? DB04 : DB20), K29XOR(( 256 & SALT) ? DB05 : DB21), K14XOR(( 512 & SALT) ? DB06 : DB22), K16XOR((1024 & SALT) ? DB07 : DB23), K50XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K21XOR(                DB23       ), K43XOR(                DB24       ), K03XOR(                DB25       ), K51XOR(                DB26       ), K09XOR(                DB27       ), K15XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K07XOR(                DB27       ), K35XOR(                DB28       ), K36XOR(                DB29       ), K24XOR(                DB30       ), K44XOR(                DB31       ), K23XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(432);
		if (!threadIdx.y) {
		s1(K27XOR((   1 & SALT) ? DB47 : DB63), K06XOR((   2 & SALT) ? DB48 : DB32), K48XOR((   4 & SALT) ? DB49 : DB33), K12XOR((   8 & SALT) ? DB50 : DB34), K32XOR((  16 & SALT) ? DB51 : DB35), K04XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K18XOR((  64 & SALT) ? DB51 : DB35), K39XOR(( 128 & SALT) ? DB52 : DB36), K20XOR(( 256 & SALT) ? DB53 : DB37), K52XOR(( 512 & SALT) ? DB54 : DB38), K33XOR((1024 & SALT) ? DB55 : DB39), K55XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K19XOR(DB39), K47XOR(DB40), K41XOR(DB41), K11XOR(DB42), K53XOR(DB43), K38XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K13XOR(DB43), K45XOR(DB44), K46XOR(DB45), K40XOR(DB46), K34XOR(DB47), K25XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K28XOR((   1 & SALT) ? DB63 : DB47), K08XOR((   2 & SALT) ? DB32 : DB48), K16XOR((   4 & SALT) ? DB33 : DB49), K03XOR((   8 & SALT) ? DB34 : DB50), K43XOR((  16 & SALT) ? DB35 : DB51), K44XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K23XOR((  64 & SALT) ? DB35 : DB51), K35XOR(( 128 & SALT) ? DB36 : DB52), K15XOR(( 256 & SALT) ? DB37 : DB53), K00XOR(( 512 & SALT) ? DB38 : DB54), K02XOR((1024 & SALT) ? DB39 : DB55), K36XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K07XOR(DB55), K29XOR(DB56), K42XOR(DB57), K37XOR(DB58), K24XOR(DB59), K01XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K50XOR(DB59), K21XOR(DB60), K22XOR(DB61), K10XOR(DB62), K30XOR(DB63), K09XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(528);
		if (!threadIdx.y) {
		s1(K13XOR((   1 & SALT) ? DB15 : DB31), K47XOR((   2 & SALT) ? DB16 : DB00), K34XOR((   4 & SALT) ? DB17 : DB01), K53XOR((   8 & SALT) ? DB18 : DB02), K18XOR((  16 & SALT) ? DB19 : DB03), K45XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K04XOR((  64 & SALT) ? DB19 : DB03), K25XOR(( 128 & SALT) ? DB20 : DB04), K06XOR(( 256 & SALT) ? DB21 : DB05), K38XOR(( 512 & SALT) ? DB22 : DB06), K19XOR((1024 & SALT) ? DB23 : DB07), K41XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K05XOR(                DB07       ), K33XOR(                DB08       ), K27XOR(                DB09       ), K52XOR(                DB10       ), K39XOR(                DB11       ), K55XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K54XOR(                DB11       ), K31XOR(                DB12       ), K32XOR(                DB13       ), K26XOR(                DB14       ), K20XOR(                DB15       ), K11XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K14XOR((   1 & SALT) ? DB31 : DB15), K51XOR((   2 & SALT) ? DB00 : DB16), K02XOR((   4 & SALT) ? DB01 : DB17), K42XOR((   8 & SALT) ? DB02 : DB18), K29XOR((  16 & SALT) ? DB03 : DB19), K30XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K09XOR((  64 & SALT) ? DB03 : DB19), K21XOR(( 128 & SALT) ? DB04 : DB20), K01XOR(( 256 & SALT) ? DB05 : DB21), K43XOR(( 512 & SALT) ? DB06 : DB22), K17XOR((1024 & SALT) ? DB07 : DB23), K22XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K50XOR(                DB23       ), K15XOR(                DB24       ), K28XOR(                DB25       ), K23XOR(                DB26       ), K10XOR(                DB27       ), K44XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K36XOR(                DB27       ), K07XOR(                DB28       ), K08XOR(                DB29       ), K49XOR(                DB30       ), K16XOR(                DB31       ), K24XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(528);
		if (!threadIdx.y) {
		s1(K54XOR((   1 & SALT) ? DB47 : DB63), K33XOR((   2 & SALT) ? DB48 : DB32), K20XOR((   4 & SALT) ? DB49 : DB33), K39XOR((   8 & SALT) ? DB50 : DB34), K04XOR((  16 & SALT) ? DB51 : DB35), K31XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K45XOR((  64 & SALT) ? DB51 : DB35), K11XOR(( 128 & SALT) ? DB52 : DB36), K47XOR(( 256 & SALT) ? DB53 : DB37), K55XOR(( 512 & SALT) ? DB54 : DB38), K05XOR((1024 & SALT) ? DB55 : DB39), K27XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K46XOR(DB39), K19XOR(DB40), K13XOR(DB41), K38XOR(DB42), K25XOR(DB43), K41XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K40XOR(DB43), K48XOR(DB44), K18XOR(DB45), K12XOR(DB46), K06XOR(DB47), K52XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K00XOR((   1 & SALT) ? DB63 : DB47), K37XOR((   2 & SALT) ? DB32 : DB48), K17XOR((   4 & SALT) ? DB33 : DB49), K28XOR((   8 & SALT) ? DB34 : DB50), K15XOR((  16 & SALT) ? DB35 : DB51), K16XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K24XOR((  64 & SALT) ? DB35 : DB51), K07XOR(( 128 & SALT) ? DB36 : DB52), K44XOR(( 256 & SALT) ? DB37 : DB53), K29XOR(( 512 & SALT) ? DB38 : DB54), K03XOR((1024 & SALT) ? DB39 : DB55), K08XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K36XOR(DB55), K01XOR(DB56), K14XOR(DB57), K09XOR(DB58), K49XOR(DB59), K30XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K22XOR(DB59), K50XOR(DB60), K51XOR(DB61), K35XOR(DB62), K02XOR(DB63), K10XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(624);
		if (!threadIdx.y) {
		s1(K40XOR((   1 & SALT) ? DB15 : DB31), K19XOR((   2 & SALT) ? DB16 : DB00), K06XOR((   4 & SALT) ? DB17 : DB01), K25XOR((   8 & SALT) ? DB18 : DB02), K45XOR((  16 & SALT) ? DB19 : DB03), K48XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K31XOR((  64 & SALT) ? DB19 : DB03), K52XOR(( 128 & SALT) ? DB20 : DB04), K33XOR(( 256 & SALT) ? DB21 : DB05), K41XOR(( 512 & SALT) ? DB22 : DB06), K46XOR((1024 & SALT) ? DB23 : DB07), K13XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K32XOR(                DB07       ), K05XOR(                DB08       ), K54XOR(                DB09       ), K55XOR(                DB10       ), K11XOR(                DB11       ), K27XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K26XOR(                DB11       ), K34XOR(                DB12       ), K04XOR(                DB13       ), K53XOR(                DB14       ), K47XOR(                DB15       ), K38XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K43XOR((   1 & SALT) ? DB31 : DB15), K23XOR((   2 & SALT) ? DB00 : DB16), K03XOR((   4 & SALT) ? DB01 : DB17), K14XOR((   8 & SALT) ? DB02 : DB18), K01XOR((  16 & SALT) ? DB03 : DB19), K02XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K10XOR((  64 & SALT) ? DB03 : DB19), K50XOR(( 128 & SALT) ? DB04 : DB20), K30XOR(( 256 & SALT) ? DB05 : DB21), K15XOR(( 512 & SALT) ? DB06 : DB22), K42XOR((1024 & SALT) ? DB07 : DB23), K51XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K22XOR(                DB23       ), K44XOR(                DB24       ), K00XOR(                DB25       ), K24XOR(                DB26       ), K35XOR(                DB27       ), K16XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K08XOR(                DB27       ), K36XOR(                DB28       ), K37XOR(                DB29       ), K21XOR(                DB30       ), K17XOR(                DB31       ), K49XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		

		// ROUND_B(624);
		if (!threadIdx.y) {
		s1(K26XOR((   1 & SALT) ? DB47 : DB63), K05XOR((   2 & SALT) ? DB48 : DB32), K47XOR((   4 & SALT) ? DB49 : DB33), K11XOR((   8 & SALT) ? DB50 : DB34), K31XOR((  16 & SALT) ? DB51 : DB35), K34XOR((  32 & SALT) ? DB52 : DB36), &DB08, &DB16, &DB22, &DB30);
		s2(K48XOR((  64 & SALT) ? DB51 : DB35), K38XOR(( 128 & SALT) ? DB52 : DB36), K19XOR(( 256 & SALT) ? DB53 : DB37), K27XOR(( 512 & SALT) ? DB54 : DB38), K32XOR((1024 & SALT) ? DB55 : DB39), K54XOR((2048 & SALT) ? DB56 : DB40), &DB12, &DB27, &DB01, &DB17);
		s3(K18XOR(DB39), K46XOR(DB40), K40XOR(DB41), K41XOR(DB42), K52XOR(DB43), K13XOR(DB44), &DB23, &DB15, &DB29, &DB05);
		s4(K12XOR(DB43), K20XOR(DB44), K45XOR(DB45), K39XOR(DB46), K33XOR(DB47), K55XOR(DB48), &DB25, &DB19, &DB09, &DB00);
		s5(K29XOR((   1 & SALT) ? DB63 : DB47), K09XOR((   2 & SALT) ? DB32 : DB48), K42XOR((   4 & SALT) ? DB33 : DB49), K00XOR((   8 & SALT) ? DB34 : DB50), K44XOR((  16 & SALT) ? DB35 : DB51), K17XOR((  32 & SALT) ? DB36 : DB52), &DB07, &DB13, &DB24, &DB02);
		s6(K49XOR((  64 & SALT) ? DB35 : DB51), K36XOR(( 128 & SALT) ? DB36 : DB52), K16XOR(( 256 & SALT) ? DB37 : DB53), K01XOR(( 512 & SALT) ? DB38 : DB54), K28XOR((1024 & SALT) ? DB39 : DB55), K37XOR((2048 & SALT) ? DB40 : DB56), &DB03, &DB28, &DB10, &DB18);
		s7(K08XOR(DB55), K30XOR(DB56), K43XOR(DB57), K10XOR(DB58), K21XOR(DB59), K02XOR(DB60), &DB31, &DB11, &DB21, &DB06);
		s8(K51XOR(DB59), K22XOR(DB60), K23XOR(DB61), K07XOR(DB62), K03XOR(DB63), K35XOR(DB32), &DB04, &DB26, &DB14, &DB20);
		}
		

		// ROUND_A(720);
		if (!threadIdx.y) {
		s1(K19XOR((   1 & SALT) ? DB15 : DB31), K53XOR((   2 & SALT) ? DB16 : DB00), K40XOR((   4 & SALT) ? DB17 : DB01), K04XOR((   8 & SALT) ? DB18 : DB02), K55XOR((  16 & SALT) ? DB19 : DB03), K27XOR((  32 & SALT) ? DB20 : DB04), &DB40, &DB48, &DB54, &DB62);
		s2(K41XOR((  64 & SALT) ? DB19 : DB03), K31XOR(( 128 & SALT) ? DB20 : DB04), K12XOR(( 256 & SALT) ? DB21 : DB05), K20XOR(( 512 & SALT) ? DB22 : DB06), K25XOR((1024 & SALT) ? DB23 : DB07), K47XOR((2048 & SALT) ? DB24 : DB08), &DB44, &DB59, &DB33, &DB49);
		s3(K11XOR(                DB07       ), K39XOR(                DB08       ), K33XOR(                DB09       ), K34XOR(                DB10       ), K45XOR(                DB11       ), K06XOR(                DB12       ), &DB55, &DB47, &DB61, &DB37);
		s4(K05XOR(                DB11       ), K13XOR(                DB12       ), K38XOR(                DB13       ), K32XOR(                DB14       ), K26XOR(                DB15       ), K48XOR(                DB16       ), &DB57, &DB51, &DB41, &DB32);
		s5(K22XOR((   1 & SALT) ? DB31 : DB15), K02XOR((   2 & SALT) ? DB00 : DB16), K35XOR((   4 & SALT) ? DB01 : DB17), K50XOR((   8 & SALT) ? DB02 : DB18), K37XOR((  16 & SALT) ? DB03 : DB19), K10XOR((  32 & SALT) ? DB04 : DB20), &DB39, &DB45, &DB56, &DB34);
		s6(K42XOR((  64 & SALT) ? DB03 : DB19), K29XOR(( 128 & SALT) ? DB04 : DB20), K09XOR(( 256 & SALT) ? DB05 : DB21), K51XOR(( 512 & SALT) ? DB06 : DB22), K21XOR((1024 & SALT) ? DB07 : DB23), K30XOR((2048 & SALT) ? DB08 : DB24), &DB35, &DB60, &DB42, &DB50);
		s7(K01XOR(                DB23       ), K23XOR(                DB24       ), K36XOR(                DB25       ), K03XOR(                DB26       ), K14XOR(                DB27       ), K24XOR(                DB28       ), &DB63, &DB43, &DB53, &DB38);
		s8(K44XOR(                DB27       ), K15XOR(                DB28       ), K16XOR(                DB29       ), K00XOR(                DB30       ), K49XOR(                DB31       ), K28XOR(                DB00       ), &DB36, &DB58, &DB46, &DB52);
		}
		
	}
	db[ 0<<DB_SHIFT] = DB00;
	db[ 1<<DB_SHIFT] = DB01;
	db[ 2<<DB_SHIFT] = DB02;
	db[ 3<<DB_SHIFT] = DB03;
	db[ 4<<DB_SHIFT] = DB04;
	db[ 5<<DB_SHIFT] = DB05;
	db[ 6<<DB_SHIFT] = DB06;
	db[ 7<<DB_SHIFT] = DB07;
	db[ 8<<DB_SHIFT] = DB08;
	db[ 9<<DB_SHIFT] = DB09;
	db[10<<DB_SHIFT] = DB10;
	db[11<<DB_SHIFT] = DB11;
	db[12<<DB_SHIFT] = DB12;
	db[13<<DB_SHIFT] = DB13;
	db[14<<DB_SHIFT] = DB14;
	db[15<<DB_SHIFT] = DB15;
	// db[16<<DB_SHIFT] = DB16;
	// db[17<<DB_SHIFT] = DB17;
	// db[18<<DB_SHIFT] = DB18;
	// db[19<<DB_SHIFT] = DB19;
	// db[20<<DB_SHIFT] = DB20;
	// db[21<<DB_SHIFT] = DB21;
	// db[22<<DB_SHIFT] = DB22;
	// db[23<<DB_SHIFT] = DB23;
	// db[24<<DB_SHIFT] = DB24;
	// db[25<<DB_SHIFT] = DB25;
	// db[26<<DB_SHIFT] = DB26;
	// db[27<<DB_SHIFT] = DB27;
	// db[28<<DB_SHIFT] = DB28;
	// db[29<<DB_SHIFT] = DB29;
	// db[30<<DB_SHIFT] = DB30;
	// db[31<<DB_SHIFT] = DB31;
	// db[32<<DB_SHIFT] = DB32;
	// db[33<<DB_SHIFT] = DB33;
	// db[34<<DB_SHIFT] = DB34;
	// db[35<<DB_SHIFT] = DB35;
	// db[36<<DB_SHIFT] = DB36;
	// db[37<<DB_SHIFT] = DB37;
	// db[38<<DB_SHIFT] = DB38;
	// db[39<<DB_SHIFT] = DB39;
	// db[40<<DB_SHIFT] = DB40;
	// db[41<<DB_SHIFT] = DB41;
	// db[42<<DB_SHIFT] = DB42;
	// db[43<<DB_SHIFT] = DB43;
	// db[44<<DB_SHIFT] = DB44;
	// db[45<<DB_SHIFT] = DB45;
	// db[46<<DB_SHIFT] = DB46;
	// db[47<<DB_SHIFT] = DB47;
	db[48<<DB_SHIFT] = DB48;
	db[49<<DB_SHIFT] = DB49;
	db[50<<DB_SHIFT] = DB50;
	db[51<<DB_SHIFT] = DB51;
	db[52<<DB_SHIFT] = DB52;
	db[53<<DB_SHIFT] = DB53;
	db[54<<DB_SHIFT] = DB54;
	db[55<<DB_SHIFT] = DB55;
	db[56<<DB_SHIFT] = DB56;
	db[57<<DB_SHIFT] = DB57;
	db[58<<DB_SHIFT] = DB58;
	db[59<<DB_SHIFT] = DB59;
	db[60<<DB_SHIFT] = DB60;
	db[61<<DB_SHIFT] = DB61;
	db[62<<DB_SHIFT] = DB62;
	db[63<<DB_SHIFT] = DB63;
}

#define GET_TRIPCODE_CHAR_INDEX(r, t, i0, i1, i2, i3, i4, i5, pos)  \
		(  ((((r)[threadIdx.x + (i0<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (5 + ((pos) * 6)))  \
	 	 | ((((r)[threadIdx.x + (i1<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (4 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i2<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (3 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i3<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (2 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i4<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (1 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i5<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << (0 + ((pos) * 6)))) \

#define GET_TRIPCODE_CHAR_INDEX_LAST(r, t, i0, i1, i2, i3)     \
		(  ((((r)[threadIdx.x + (i0<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 5)  \
	 	 | ((((r)[threadIdx.x + (i1<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 4)  \
		 | ((((r)[threadIdx.x + (i2<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 3)  \
		 | ((((r)[threadIdx.x + (i3<<DB_SHIFT)] & (0x01 << (t))) ? (0x1) : (0x0)) << 2)) \

DES_FUNCTION_QUALIFIERS void
DES_GetTripcodeChunks(int tripcodeIndex, unsigned int *tripcodeChunkArray, int searchMode)
{
	// Perform the final permutation here.
	if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
	} else if (searchMode == SEARCH_MODE_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
		                        | GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
								| GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] = ((tripcodeChunkArray[0] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0);
		tripcodeChunkArray[2] = ((tripcodeChunkArray[1] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0);
		tripcodeChunkArray[3] = ((tripcodeChunkArray[2] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0);
		tripcodeChunkArray[4] = ((tripcodeChunkArray[3] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0);
		tripcodeChunkArray[5] = ((tripcodeChunkArray[4] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	}
}

DES_FUNCTION_QUALIFIERS
unsigned char *DES_GetTripcode(int tripcodeIndex, unsigned char *tripcode)
{
	// Perform the final permutation as necessary.
  	tripcode[0] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0)];
  	tripcode[1] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0)];
  	tripcode[2] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0)];
  	tripcode[3] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0)];
  	tripcode[4] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0)];
  	tripcode[5] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0)];
  	tripcode[6] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0)];
  	tripcode[7] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0)];
  	tripcode[8] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0)];
	tripcode[9] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24)];
 	tripcode[10] = '\0';

	return tripcode;
}

#define SET_KEY_CHAR(var, flag, table, value)             \
	if (!(flag)) {                                        \
		var = (table)[(value)];                           \
		isSecondByte = IS_FIRST_BYTE_SJIS(var);           \
	} else {                                              \
		var = CUDA_keyCharTable_SecondByte[(value)];          \
		isSecondByte = FALSE;                             \
	}

#define CUDA_DES_DEFINE_SEARCH_FUNCTION(functionName) \
__global__ void functionName(\
	GPUOutput *outputArray,\
	unsigned char      *keyBitmap,\
	unsigned int     *tripcodeChunkArray,\
	unsigned int      numTripcodeChunk,\
	unsigned int keyFrom00To27,\
	int          intSalt,\
	int         searchMode) {

#define CUDA_DES_BEFORE_SEARCHING \
	GPUOutput  *output = &outputArray[blockIdx.x * CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK + threadIdx.x];\
	unsigned char        key[8];\
	BOOL         isSecondByte;\
	unsigned char        tripcodeIndex;\
	unsigned char        passCount;\
	\
	if (threadIdx.y == 0) {\
		output->numMatchingTripcodes = 0;\
	}\
	key[0] = CUDA_key[0];\
	key[1] = CUDA_key[1];\
	key[2] = CUDA_key[2];\
	key[3] = CUDA_key[3];\
	isSecondByte = IS_FIRST_BYTE_SJIS(CUDA_key[3]);\
	SET_KEY_CHAR(key[4], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[4] + ((blockIdx.x  >> 6) & 63));\
	SET_KEY_CHAR(key[5], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[5] + ( blockIdx.x        & 63));\
	SET_KEY_CHAR(key[6], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[6] + ( threadIdx.x       & 63));\
	unsigned int keyFrom28To48 = (((unsigned int)key[6] & 0x7f) << 14) | (((unsigned int)key[5] & 0x7f) <<  7) | (((unsigned int)key[4] & 0x7f) << 0); \
	\
	for (passCount = 0; passCount < 1; ++passCount) {\
	__syncthreads();\
	DES_Crypt(keyFrom00To27, keyFrom28To48, intSalt);\
	\
	__syncthreads();\
	if (threadIdx.y == 0) {\
		BOOL found = FALSE;\
		for (tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {

#define CUDA_DES_END_OF_SEAERCH_FUNCTION \
		}\
quit_loops:\
		if (found == TRUE) {\
			output->numMatchingTripcodes  = 1;\
			output->pair.key.c[0] = key[0];\
			output->pair.key.c[1] = key[1];\
			output->pair.key.c[2] = key[2];\
			output->pair.key.c[3] = key[3];\
			output->pair.key.c[4] = key[4];\
			output->pair.key.c[5] = key[5];\
			output->pair.key.c[6] = key[6];\
			output->pair.key.c[7] = CUDA_key7Array[tripcodeIndex];\
		}\
	}\
	}\
	if (threadIdx.y == 0)\
		output->numGeneratedTripcodes = CUDA_DES_BS_DEPTH;\
}

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == tripcodeChunk) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	if (keyBitmap[tripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)])
		continue;
	for (int j = 0; j < numTripcodeChunk; ++j){
		int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
		while (tripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
			middle = (lower + upper) >> 1;
			if (tripcodeChunk > tripcodeChunkArray[middle]) {
				lower = middle + 1;
			} else {
				upper = middle - 1;
			}
		}
		if (tripcodeChunk == tripcodeChunkArray[middle]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_BackwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[pos]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		unsigned int generatedTripcodeChunk = generatedTripcodeChunkArray[pos];
		if (keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)])
			continue;
		for (int j = 0; j < numTripcodeChunk; ++j){
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

/*
CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[0]) {
			found = TRUE;
			goto quit_loops;
		}
	}
	//
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[1]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION
*/

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching)
	unsigned int generatedTripcodeChunkArray[6];
	unsigned int generatedTripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[0];
	if (!keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[1];
	if (!keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j) {
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION



///////////////////////////////////////////////////////////////////////////////
// CUDA SEARCH THREAD FOR 10 CHARACTER TRIPCODES                             //
///////////////////////////////////////////////////////////////////////////////

#define SET_BIT_FOR_KEY7(var, k) if (key7 & (0x1 << (k))) (var) |= 0x1 << tripcodeIndex

unsigned WINAPI Thread_SearchForDESTripcodesOnCUDADevice(LPVOID info)
{
	hipError_t     hipError_t;
	hipDeviceProp_t  CUDADeviceProperties;
	unsigned int    numBlocksPerSM;
	unsigned int    numBlocksPerGrid;
	GPUOutput      *outputArray = NULL;
	GPUOutput      *CUDA_outputArray = NULL;
	unsigned int   *CUDA_tripcodeChunkArray = NULL;
	unsigned char  *CUDA_keyBitmap = NULL;
	unsigned int    sizeOutputArray;
	unsigned char   key[MAX_LEN_TRIPCODE + 1];
	unsigned char   salt[3];
	unsigned char   expansionFunction[96];
	char            status[LEN_LINE_BUFFER_FOR_SCREEN] = "";
	int             optimizationPhase    = CUDA_OPTIMIZATION_PHASE_NUM_BLOCKS;
	int             optimizationSubphase = 0;
	double          timeElapsedInOptimizationSubphase = 0;
	static int      numBlocksTableForOptimization[] = {8, 16, 32, 48, 64, 96, 128, 160, 192, 224, 256, 0};
	double          numGeneratedTripcodes = 0;
	double          speedInPreviousSubphase = 0;
	double          speedInCurrentSubphase = 0;
	DWORD           startingTime;
	DWORD           endingTime;
	double          deltaTime;

	key[lenTripcode] = '\0';
	salt[2]          = '\0';
	
	CUDA_ERROR(hipSetDevice(((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	CUDA_ERROR(hipGetDeviceProperties(&CUDADeviceProperties, ((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	if (CUDADeviceProperties.computeMode == hipComputeModeProhibited) {
		sprintf(status, "[disabled]");
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), FALSE, status);
		return 0;
	}

	if (options.CUDANumBlocksPerSM == CUDA_NUM_BLOCKS_PER_SM_NIL) {
		numBlocksPerSM = numBlocksTableForOptimization[optimizationSubphase];
	} else {
		numBlocksPerSM = options.CUDANumBlocksPerSM;
	}
	numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
	sizeOutputArray = CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK * numBlocksPerGrid;
	outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(outputArray == NULL, ERROR_NO_MEMORY, "Not enough memory.");
	hipError_t = hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);
	hipError_t = hipMalloc((void **)&CUDA_keyBitmap, KEY_BITMAP_SIZE);
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);
	hipError_t = hipMalloc((void **)&CUDA_tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk); 
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);

	CUDA_ERROR(hipMemcpy(CUDA_tripcodeChunkArray, tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpy(CUDA_keyBitmap, keyBitmap, KEY_BITMAP_SIZE, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	
	startingTime = timeGetTime();

	while (!GetTerminationState()) {
		// Choose the first 4 characters of the key.
		SetCharactersInTripcodeKey(key, 4);
		
		// Make sure that the first 3 bytes consist of valid Shift-JIS characters.
		for (int i = 4; i < lenTripcode; ++i)
			key[i] = 'A';
		if (!IsValidKey(key))
			continue;
		unsigned int keyFrom00To27 = (((unsigned int)key[3] & 0x7f) << 21) | (((unsigned int)key[2] & 0x7f) << 14) | (((unsigned int)key[1] & 0x7f) << 7) | (((unsigned int)key[0] & 0x7f) << 0); \
		
		// Generate random bytes for the key to ensure the randomness of them.
		unsigned char randomByteForKey6 = RandomByte();
		for (int i = 4; i < lenTripcode; ++i)
			key[i] = RandomByte();
		unsigned char key7Array[CUDA_DES_BS_DEPTH];
		DES_Vector  keyFrom49To55Array[7] = {0, 0, 0, 0, 0, 0, 0};
		for (int tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {
			unsigned char key7 = key7Array[tripcodeIndex] = keyCharTable_SecondByteAndOneByte[key[7] + tripcodeIndex];
			SET_BIT_FOR_KEY7(keyFrom49To55Array[0], 0);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[1], 1);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[2], 2);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[3], 3);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[4], 4);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[5], 5);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[6], 6);
		}

		// Create an expansion function based on the salt.
		salt[0] = CONVERT_CHAR_FOR_SALT(key[1]);
		salt[1] = CONVERT_CHAR_FOR_SALT(key[2]);
		int intSalt = charToIndexTableForDES[CONVERT_CHAR_FOR_SALT(key[1])] | (charToIndexTableForDES[CONVERT_CHAR_FOR_SALT(key[2])] << 6);
		DES_CreateExpansionFunction((char *)salt, expansionFunction);

		// Call an appropriate CUDA kernel.
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcode));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(expansionFunction)));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(key7Array)));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(keyFrom49To55Array)));
		dim3 dimBlock(CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK, CUDA_DES_NUM_THREADS_FOR_BITSLICE);
		dim3 dimGrid(numBlocksPerGrid);
		if (searchMode == SEARCH_MODE_FLEXIBLE) {
			/*if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_Flexible_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			} else {*/
				CUDA_PerformSearching_DES_Flexible<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			//}
		} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
			/*if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			} else {*/
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			//}
		} else {
			/*
			if (numTripcodeChunk == 1) {
				if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
					CUDA_PerformSearching_DES_ForwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
						CUDA_outputArray,
						CUDA_keyBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						keyFrom00To27,
						intSalt,
						searchMode);
				} else {
					CUDA_PerformSearching_DES_BackwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
						CUDA_outputArray,
						CUDA_keyBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						keyFrom00To27,
						intSalt,
						searchMode);
				}
			} else if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			} else {*/
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					keyFrom00To27,
					intSalt,
					searchMode);
			//}
		}
		CUDA_ERROR(hipGetLastError());
		// CUDA_ERROR(hipDeviceSynchronize()); // Check errors at kernel launch.

		// Process the output array.
		CUDA_ERROR(hipMemcpy(outputArray, CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray, hipMemcpyDeviceToHost));
		// We can save registers this way. Not particularly safe, though.
		for (unsigned int indexOutput = 0; indexOutput < sizeOutputArray; indexOutput++){
			GPUOutput *output = &outputArray[indexOutput];
			if (output->numMatchingTripcodes > 0)
				GenerateDESTripcode(output->pair.tripcode.c, output->pair.key.c);
		}
		numGeneratedTripcodes += ProcessGPUOutput(key, outputArray, sizeOutputArray, FALSE);
		
		// Optimization
		endingTime = timeGetTime();
		deltaTime = (endingTime >= startingTime)
						? ((double)endingTime - (double)startingTime                     ) * 0.001
						: ((double)endingTime - (double)startingTime + (double)0xffffffff) * 0.001;
		while (GetPauseState() && !GetTerminationState())
			Sleep(PAUSE_INTERVAL);
		startingTime = timeGetTime();
		timeElapsedInOptimizationSubphase += deltaTime;
		speedInCurrentSubphase = numGeneratedTripcodes / timeElapsedInOptimizationSubphase;
		//
		if (optimizationPhase == CUDA_OPTIMIZATION_PHASE_NUM_BLOCKS) {
			if (options.CUDANumBlocksPerSM != CUDA_NUM_BLOCKS_PER_SM_NIL) {
				optimizationPhase     = CUDA_OPTIMIZATION_PHASE_COMPLETED;
				optimizationSubphase  = 0;
				numGeneratedTripcodes = 0;
				timeElapsedInOptimizationSubphase = 0;
			} else if (timeElapsedInOptimizationSubphase >= CUDA_OPTIMIZATION_SUBPHASE_DURATION) {
				if (   optimizationSubphase > 0
				    && (   speedInPreviousSubphase > speedInCurrentSubphase
					    || fabs(speedInPreviousSubphase - speedInCurrentSubphase) / speedInPreviousSubphase < CUDA_OPTIMIZATION_THRESHOLD)) {
					numBlocksPerSM = numBlocksTableForOptimization[(speedInPreviousSubphase > speedInCurrentSubphase) ? (optimizationSubphase - 1) : (optimizationSubphase)];
					optimizationPhase = CUDA_OPTIMIZATION_PHASE_COMPLETED;
					optimizationSubphase = 0;
					numGeneratedTripcodes = 0;
				} else if (numBlocksTableForOptimization[optimizationSubphase + 1] > 0) {
					numBlocksPerSM = numBlocksTableForOptimization[++optimizationSubphase];
					timeElapsedInOptimizationSubphase = 0;
					numGeneratedTripcodes = 0;
					speedInPreviousSubphase = speedInCurrentSubphase;
				} else {
					optimizationPhase = CUDA_OPTIMIZATION_PHASE_COMPLETED;
					optimizationSubphase = 0;
					numGeneratedTripcodes = 0;
				}
				timeElapsedInOptimizationSubphase = 0;
				numGeneratedTripcodes = 0;
				numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
				sizeOutputArray = CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK * numBlocksPerGrid;
				free(outputArray);
				outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
				ERROR0(outputArray == NULL, ERROR_NO_MEMORY, "Not enough memory.");
				CUDA_ERROR(hipFree(CUDA_outputArray));
				hipError_t = hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray);
				ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
				CUDA_ERROR(hipError_t);
			}
		}
		//
		sprintf(status,
			    "%s%.1lfM TPS, %d blocks/SM",
				(optimizationPhase != CUDA_OPTIMIZATION_PHASE_COMPLETED) ? "[optimizing...] " : "",
				speedInCurrentSubphase / 1000000,
				numBlocksPerSM);
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), (optimizationPhase != CUDA_OPTIMIZATION_PHASE_COMPLETED), status);
	}

	RELEASE_AND_SET_TO_NULL(CUDA_outputArray,        hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_tripcodeChunkArray, hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_keyBitmap,          hipFree);
	RELEASE_AND_SET_TO_NULL(outputArray,             free);
}
