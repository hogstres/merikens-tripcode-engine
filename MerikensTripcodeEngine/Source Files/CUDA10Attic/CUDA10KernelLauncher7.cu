// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 3584
#include "CUDA10Kernel.h"
#define SALT 3585
#include "CUDA10Kernel.h"
#define SALT 3586
#include "CUDA10Kernel.h"
#define SALT 3587
#include "CUDA10Kernel.h"
#define SALT 3588
#include "CUDA10Kernel.h"
#define SALT 3589
#include "CUDA10Kernel.h"
#define SALT 3590
#include "CUDA10Kernel.h"
#define SALT 3591
#include "CUDA10Kernel.h"
#define SALT 3592
#include "CUDA10Kernel.h"
#define SALT 3593
#include "CUDA10Kernel.h"
#define SALT 3594
#include "CUDA10Kernel.h"
#define SALT 3595
#include "CUDA10Kernel.h"
#define SALT 3596
#include "CUDA10Kernel.h"
#define SALT 3597
#include "CUDA10Kernel.h"
#define SALT 3598
#include "CUDA10Kernel.h"
#define SALT 3599
#include "CUDA10Kernel.h"
#define SALT 3600
#include "CUDA10Kernel.h"
#define SALT 3601
#include "CUDA10Kernel.h"
#define SALT 3602
#include "CUDA10Kernel.h"
#define SALT 3603
#include "CUDA10Kernel.h"
#define SALT 3604
#include "CUDA10Kernel.h"
#define SALT 3605
#include "CUDA10Kernel.h"
#define SALT 3606
#include "CUDA10Kernel.h"
#define SALT 3607
#include "CUDA10Kernel.h"
#define SALT 3608
#include "CUDA10Kernel.h"
#define SALT 3609
#include "CUDA10Kernel.h"
#define SALT 3610
#include "CUDA10Kernel.h"
#define SALT 3611
#include "CUDA10Kernel.h"
#define SALT 3612
#include "CUDA10Kernel.h"
#define SALT 3613
#include "CUDA10Kernel.h"
#define SALT 3614
#include "CUDA10Kernel.h"
#define SALT 3615
#include "CUDA10Kernel.h"
#define SALT 3616
#include "CUDA10Kernel.h"
#define SALT 3617
#include "CUDA10Kernel.h"
#define SALT 3618
#include "CUDA10Kernel.h"
#define SALT 3619
#include "CUDA10Kernel.h"
#define SALT 3620
#include "CUDA10Kernel.h"
#define SALT 3621
#include "CUDA10Kernel.h"
#define SALT 3622
#include "CUDA10Kernel.h"
#define SALT 3623
#include "CUDA10Kernel.h"
#define SALT 3624
#include "CUDA10Kernel.h"
#define SALT 3625
#include "CUDA10Kernel.h"
#define SALT 3626
#include "CUDA10Kernel.h"
#define SALT 3627
#include "CUDA10Kernel.h"
#define SALT 3628
#include "CUDA10Kernel.h"
#define SALT 3629
#include "CUDA10Kernel.h"
#define SALT 3630
#include "CUDA10Kernel.h"
#define SALT 3631
#include "CUDA10Kernel.h"
#define SALT 3632
#include "CUDA10Kernel.h"
#define SALT 3633
#include "CUDA10Kernel.h"
#define SALT 3634
#include "CUDA10Kernel.h"
#define SALT 3635
#include "CUDA10Kernel.h"
#define SALT 3636
#include "CUDA10Kernel.h"
#define SALT 3637
#include "CUDA10Kernel.h"
#define SALT 3638
#include "CUDA10Kernel.h"
#define SALT 3639
#include "CUDA10Kernel.h"
#define SALT 3640
#include "CUDA10Kernel.h"
#define SALT 3641
#include "CUDA10Kernel.h"
#define SALT 3642
#include "CUDA10Kernel.h"
#define SALT 3643
#include "CUDA10Kernel.h"
#define SALT 3644
#include "CUDA10Kernel.h"
#define SALT 3645
#include "CUDA10Kernel.h"
#define SALT 3646
#include "CUDA10Kernel.h"
#define SALT 3647
#include "CUDA10Kernel.h"
#define SALT 3648
#include "CUDA10Kernel.h"
#define SALT 3649
#include "CUDA10Kernel.h"
#define SALT 3650
#include "CUDA10Kernel.h"
#define SALT 3651
#include "CUDA10Kernel.h"
#define SALT 3652
#include "CUDA10Kernel.h"
#define SALT 3653
#include "CUDA10Kernel.h"
#define SALT 3654
#include "CUDA10Kernel.h"
#define SALT 3655
#include "CUDA10Kernel.h"
#define SALT 3656
#include "CUDA10Kernel.h"
#define SALT 3657
#include "CUDA10Kernel.h"
#define SALT 3658
#include "CUDA10Kernel.h"
#define SALT 3659
#include "CUDA10Kernel.h"
#define SALT 3660
#include "CUDA10Kernel.h"
#define SALT 3661
#include "CUDA10Kernel.h"
#define SALT 3662
#include "CUDA10Kernel.h"
#define SALT 3663
#include "CUDA10Kernel.h"
#define SALT 3664
#include "CUDA10Kernel.h"
#define SALT 3665
#include "CUDA10Kernel.h"
#define SALT 3666
#include "CUDA10Kernel.h"
#define SALT 3667
#include "CUDA10Kernel.h"
#define SALT 3668
#include "CUDA10Kernel.h"
#define SALT 3669
#include "CUDA10Kernel.h"
#define SALT 3670
#include "CUDA10Kernel.h"
#define SALT 3671
#include "CUDA10Kernel.h"
#define SALT 3672
#include "CUDA10Kernel.h"
#define SALT 3673
#include "CUDA10Kernel.h"
#define SALT 3674
#include "CUDA10Kernel.h"
#define SALT 3675
#include "CUDA10Kernel.h"
#define SALT 3676
#include "CUDA10Kernel.h"
#define SALT 3677
#include "CUDA10Kernel.h"
#define SALT 3678
#include "CUDA10Kernel.h"
#define SALT 3679
#include "CUDA10Kernel.h"
#define SALT 3680
#include "CUDA10Kernel.h"
#define SALT 3681
#include "CUDA10Kernel.h"
#define SALT 3682
#include "CUDA10Kernel.h"
#define SALT 3683
#include "CUDA10Kernel.h"
#define SALT 3684
#include "CUDA10Kernel.h"
#define SALT 3685
#include "CUDA10Kernel.h"
#define SALT 3686
#include "CUDA10Kernel.h"
#define SALT 3687
#include "CUDA10Kernel.h"
#define SALT 3688
#include "CUDA10Kernel.h"
#define SALT 3689
#include "CUDA10Kernel.h"
#define SALT 3690
#include "CUDA10Kernel.h"
#define SALT 3691
#include "CUDA10Kernel.h"
#define SALT 3692
#include "CUDA10Kernel.h"
#define SALT 3693
#include "CUDA10Kernel.h"
#define SALT 3694
#include "CUDA10Kernel.h"
#define SALT 3695
#include "CUDA10Kernel.h"
#define SALT 3696
#include "CUDA10Kernel.h"
#define SALT 3697
#include "CUDA10Kernel.h"
#define SALT 3698
#include "CUDA10Kernel.h"
#define SALT 3699
#include "CUDA10Kernel.h"
#define SALT 3700
#include "CUDA10Kernel.h"
#define SALT 3701
#include "CUDA10Kernel.h"
#define SALT 3702
#include "CUDA10Kernel.h"
#define SALT 3703
#include "CUDA10Kernel.h"
#define SALT 3704
#include "CUDA10Kernel.h"
#define SALT 3705
#include "CUDA10Kernel.h"
#define SALT 3706
#include "CUDA10Kernel.h"
#define SALT 3707
#include "CUDA10Kernel.h"
#define SALT 3708
#include "CUDA10Kernel.h"
#define SALT 3709
#include "CUDA10Kernel.h"
#define SALT 3710
#include "CUDA10Kernel.h"
#define SALT 3711
#include "CUDA10Kernel.h"
#define SALT 3712
#include "CUDA10Kernel.h"
#define SALT 3713
#include "CUDA10Kernel.h"
#define SALT 3714
#include "CUDA10Kernel.h"
#define SALT 3715
#include "CUDA10Kernel.h"
#define SALT 3716
#include "CUDA10Kernel.h"
#define SALT 3717
#include "CUDA10Kernel.h"
#define SALT 3718
#include "CUDA10Kernel.h"
#define SALT 3719
#include "CUDA10Kernel.h"
#define SALT 3720
#include "CUDA10Kernel.h"
#define SALT 3721
#include "CUDA10Kernel.h"
#define SALT 3722
#include "CUDA10Kernel.h"
#define SALT 3723
#include "CUDA10Kernel.h"
#define SALT 3724
#include "CUDA10Kernel.h"
#define SALT 3725
#include "CUDA10Kernel.h"
#define SALT 3726
#include "CUDA10Kernel.h"
#define SALT 3727
#include "CUDA10Kernel.h"
#define SALT 3728
#include "CUDA10Kernel.h"
#define SALT 3729
#include "CUDA10Kernel.h"
#define SALT 3730
#include "CUDA10Kernel.h"
#define SALT 3731
#include "CUDA10Kernel.h"
#define SALT 3732
#include "CUDA10Kernel.h"
#define SALT 3733
#include "CUDA10Kernel.h"
#define SALT 3734
#include "CUDA10Kernel.h"
#define SALT 3735
#include "CUDA10Kernel.h"
#define SALT 3736
#include "CUDA10Kernel.h"
#define SALT 3737
#include "CUDA10Kernel.h"
#define SALT 3738
#include "CUDA10Kernel.h"
#define SALT 3739
#include "CUDA10Kernel.h"
#define SALT 3740
#include "CUDA10Kernel.h"
#define SALT 3741
#include "CUDA10Kernel.h"
#define SALT 3742
#include "CUDA10Kernel.h"
#define SALT 3743
#include "CUDA10Kernel.h"
#define SALT 3744
#include "CUDA10Kernel.h"
#define SALT 3745
#include "CUDA10Kernel.h"
#define SALT 3746
#include "CUDA10Kernel.h"
#define SALT 3747
#include "CUDA10Kernel.h"
#define SALT 3748
#include "CUDA10Kernel.h"
#define SALT 3749
#include "CUDA10Kernel.h"
#define SALT 3750
#include "CUDA10Kernel.h"
#define SALT 3751
#include "CUDA10Kernel.h"
#define SALT 3752
#include "CUDA10Kernel.h"
#define SALT 3753
#include "CUDA10Kernel.h"
#define SALT 3754
#include "CUDA10Kernel.h"
#define SALT 3755
#include "CUDA10Kernel.h"
#define SALT 3756
#include "CUDA10Kernel.h"
#define SALT 3757
#include "CUDA10Kernel.h"
#define SALT 3758
#include "CUDA10Kernel.h"
#define SALT 3759
#include "CUDA10Kernel.h"
#define SALT 3760
#include "CUDA10Kernel.h"
#define SALT 3761
#include "CUDA10Kernel.h"
#define SALT 3762
#include "CUDA10Kernel.h"
#define SALT 3763
#include "CUDA10Kernel.h"
#define SALT 3764
#include "CUDA10Kernel.h"
#define SALT 3765
#include "CUDA10Kernel.h"
#define SALT 3766
#include "CUDA10Kernel.h"
#define SALT 3767
#include "CUDA10Kernel.h"
#define SALT 3768
#include "CUDA10Kernel.h"
#define SALT 3769
#include "CUDA10Kernel.h"
#define SALT 3770
#include "CUDA10Kernel.h"
#define SALT 3771
#include "CUDA10Kernel.h"
#define SALT 3772
#include "CUDA10Kernel.h"
#define SALT 3773
#include "CUDA10Kernel.h"
#define SALT 3774
#include "CUDA10Kernel.h"
#define SALT 3775
#include "CUDA10Kernel.h"
#define SALT 3776
#include "CUDA10Kernel.h"
#define SALT 3777
#include "CUDA10Kernel.h"
#define SALT 3778
#include "CUDA10Kernel.h"
#define SALT 3779
#include "CUDA10Kernel.h"
#define SALT 3780
#include "CUDA10Kernel.h"
#define SALT 3781
#include "CUDA10Kernel.h"
#define SALT 3782
#include "CUDA10Kernel.h"
#define SALT 3783
#include "CUDA10Kernel.h"
#define SALT 3784
#include "CUDA10Kernel.h"
#define SALT 3785
#include "CUDA10Kernel.h"
#define SALT 3786
#include "CUDA10Kernel.h"
#define SALT 3787
#include "CUDA10Kernel.h"
#define SALT 3788
#include "CUDA10Kernel.h"
#define SALT 3789
#include "CUDA10Kernel.h"
#define SALT 3790
#include "CUDA10Kernel.h"
#define SALT 3791
#include "CUDA10Kernel.h"
#define SALT 3792
#include "CUDA10Kernel.h"
#define SALT 3793
#include "CUDA10Kernel.h"
#define SALT 3794
#include "CUDA10Kernel.h"
#define SALT 3795
#include "CUDA10Kernel.h"
#define SALT 3796
#include "CUDA10Kernel.h"
#define SALT 3797
#include "CUDA10Kernel.h"
#define SALT 3798
#include "CUDA10Kernel.h"
#define SALT 3799
#include "CUDA10Kernel.h"
#define SALT 3800
#include "CUDA10Kernel.h"
#define SALT 3801
#include "CUDA10Kernel.h"
#define SALT 3802
#include "CUDA10Kernel.h"
#define SALT 3803
#include "CUDA10Kernel.h"
#define SALT 3804
#include "CUDA10Kernel.h"
#define SALT 3805
#include "CUDA10Kernel.h"
#define SALT 3806
#include "CUDA10Kernel.h"
#define SALT 3807
#include "CUDA10Kernel.h"
#define SALT 3808
#include "CUDA10Kernel.h"
#define SALT 3809
#include "CUDA10Kernel.h"
#define SALT 3810
#include "CUDA10Kernel.h"
#define SALT 3811
#include "CUDA10Kernel.h"
#define SALT 3812
#include "CUDA10Kernel.h"
#define SALT 3813
#include "CUDA10Kernel.h"
#define SALT 3814
#include "CUDA10Kernel.h"
#define SALT 3815
#include "CUDA10Kernel.h"
#define SALT 3816
#include "CUDA10Kernel.h"
#define SALT 3817
#include "CUDA10Kernel.h"
#define SALT 3818
#include "CUDA10Kernel.h"
#define SALT 3819
#include "CUDA10Kernel.h"
#define SALT 3820
#include "CUDA10Kernel.h"
#define SALT 3821
#include "CUDA10Kernel.h"
#define SALT 3822
#include "CUDA10Kernel.h"
#define SALT 3823
#include "CUDA10Kernel.h"
#define SALT 3824
#include "CUDA10Kernel.h"
#define SALT 3825
#include "CUDA10Kernel.h"
#define SALT 3826
#include "CUDA10Kernel.h"
#define SALT 3827
#include "CUDA10Kernel.h"
#define SALT 3828
#include "CUDA10Kernel.h"
#define SALT 3829
#include "CUDA10Kernel.h"
#define SALT 3830
#include "CUDA10Kernel.h"
#define SALT 3831
#include "CUDA10Kernel.h"
#define SALT 3832
#include "CUDA10Kernel.h"
#define SALT 3833
#include "CUDA10Kernel.h"
#define SALT 3834
#include "CUDA10Kernel.h"
#define SALT 3835
#include "CUDA10Kernel.h"
#define SALT 3836
#include "CUDA10Kernel.h"
#define SALT 3837
#include "CUDA10Kernel.h"
#define SALT 3838
#include "CUDA10Kernel.h"
#define SALT 3839
#include "CUDA10Kernel.h"
#define SALT 3840
#include "CUDA10Kernel.h"
#define SALT 3841
#include "CUDA10Kernel.h"
#define SALT 3842
#include "CUDA10Kernel.h"
#define SALT 3843
#include "CUDA10Kernel.h"
#define SALT 3844
#include "CUDA10Kernel.h"
#define SALT 3845
#include "CUDA10Kernel.h"
#define SALT 3846
#include "CUDA10Kernel.h"
#define SALT 3847
#include "CUDA10Kernel.h"
#define SALT 3848
#include "CUDA10Kernel.h"
#define SALT 3849
#include "CUDA10Kernel.h"
#define SALT 3850
#include "CUDA10Kernel.h"
#define SALT 3851
#include "CUDA10Kernel.h"
#define SALT 3852
#include "CUDA10Kernel.h"
#define SALT 3853
#include "CUDA10Kernel.h"
#define SALT 3854
#include "CUDA10Kernel.h"
#define SALT 3855
#include "CUDA10Kernel.h"
#define SALT 3856
#include "CUDA10Kernel.h"
#define SALT 3857
#include "CUDA10Kernel.h"
#define SALT 3858
#include "CUDA10Kernel.h"
#define SALT 3859
#include "CUDA10Kernel.h"
#define SALT 3860
#include "CUDA10Kernel.h"
#define SALT 3861
#include "CUDA10Kernel.h"
#define SALT 3862
#include "CUDA10Kernel.h"
#define SALT 3863
#include "CUDA10Kernel.h"
#define SALT 3864
#include "CUDA10Kernel.h"
#define SALT 3865
#include "CUDA10Kernel.h"
#define SALT 3866
#include "CUDA10Kernel.h"
#define SALT 3867
#include "CUDA10Kernel.h"
#define SALT 3868
#include "CUDA10Kernel.h"
#define SALT 3869
#include "CUDA10Kernel.h"
#define SALT 3870
#include "CUDA10Kernel.h"
#define SALT 3871
#include "CUDA10Kernel.h"
#define SALT 3872
#include "CUDA10Kernel.h"
#define SALT 3873
#include "CUDA10Kernel.h"
#define SALT 3874
#include "CUDA10Kernel.h"
#define SALT 3875
#include "CUDA10Kernel.h"
#define SALT 3876
#include "CUDA10Kernel.h"
#define SALT 3877
#include "CUDA10Kernel.h"
#define SALT 3878
#include "CUDA10Kernel.h"
#define SALT 3879
#include "CUDA10Kernel.h"
#define SALT 3880
#include "CUDA10Kernel.h"
#define SALT 3881
#include "CUDA10Kernel.h"
#define SALT 3882
#include "CUDA10Kernel.h"
#define SALT 3883
#include "CUDA10Kernel.h"
#define SALT 3884
#include "CUDA10Kernel.h"
#define SALT 3885
#include "CUDA10Kernel.h"
#define SALT 3886
#include "CUDA10Kernel.h"
#define SALT 3887
#include "CUDA10Kernel.h"
#define SALT 3888
#include "CUDA10Kernel.h"
#define SALT 3889
#include "CUDA10Kernel.h"
#define SALT 3890
#include "CUDA10Kernel.h"
#define SALT 3891
#include "CUDA10Kernel.h"
#define SALT 3892
#include "CUDA10Kernel.h"
#define SALT 3893
#include "CUDA10Kernel.h"
#define SALT 3894
#include "CUDA10Kernel.h"
#define SALT 3895
#include "CUDA10Kernel.h"
#define SALT 3896
#include "CUDA10Kernel.h"
#define SALT 3897
#include "CUDA10Kernel.h"
#define SALT 3898
#include "CUDA10Kernel.h"
#define SALT 3899
#include "CUDA10Kernel.h"
#define SALT 3900
#include "CUDA10Kernel.h"
#define SALT 3901
#include "CUDA10Kernel.h"
#define SALT 3902
#include "CUDA10Kernel.h"
#define SALT 3903
#include "CUDA10Kernel.h"
#define SALT 3904
#include "CUDA10Kernel.h"
#define SALT 3905
#include "CUDA10Kernel.h"
#define SALT 3906
#include "CUDA10Kernel.h"
#define SALT 3907
#include "CUDA10Kernel.h"
#define SALT 3908
#include "CUDA10Kernel.h"
#define SALT 3909
#include "CUDA10Kernel.h"
#define SALT 3910
#include "CUDA10Kernel.h"
#define SALT 3911
#include "CUDA10Kernel.h"
#define SALT 3912
#include "CUDA10Kernel.h"
#define SALT 3913
#include "CUDA10Kernel.h"
#define SALT 3914
#include "CUDA10Kernel.h"
#define SALT 3915
#include "CUDA10Kernel.h"
#define SALT 3916
#include "CUDA10Kernel.h"
#define SALT 3917
#include "CUDA10Kernel.h"
#define SALT 3918
#include "CUDA10Kernel.h"
#define SALT 3919
#include "CUDA10Kernel.h"
#define SALT 3920
#include "CUDA10Kernel.h"
#define SALT 3921
#include "CUDA10Kernel.h"
#define SALT 3922
#include "CUDA10Kernel.h"
#define SALT 3923
#include "CUDA10Kernel.h"
#define SALT 3924
#include "CUDA10Kernel.h"
#define SALT 3925
#include "CUDA10Kernel.h"
#define SALT 3926
#include "CUDA10Kernel.h"
#define SALT 3927
#include "CUDA10Kernel.h"
#define SALT 3928
#include "CUDA10Kernel.h"
#define SALT 3929
#include "CUDA10Kernel.h"
#define SALT 3930
#include "CUDA10Kernel.h"
#define SALT 3931
#include "CUDA10Kernel.h"
#define SALT 3932
#include "CUDA10Kernel.h"
#define SALT 3933
#include "CUDA10Kernel.h"
#define SALT 3934
#include "CUDA10Kernel.h"
#define SALT 3935
#include "CUDA10Kernel.h"
#define SALT 3936
#include "CUDA10Kernel.h"
#define SALT 3937
#include "CUDA10Kernel.h"
#define SALT 3938
#include "CUDA10Kernel.h"
#define SALT 3939
#include "CUDA10Kernel.h"
#define SALT 3940
#include "CUDA10Kernel.h"
#define SALT 3941
#include "CUDA10Kernel.h"
#define SALT 3942
#include "CUDA10Kernel.h"
#define SALT 3943
#include "CUDA10Kernel.h"
#define SALT 3944
#include "CUDA10Kernel.h"
#define SALT 3945
#include "CUDA10Kernel.h"
#define SALT 3946
#include "CUDA10Kernel.h"
#define SALT 3947
#include "CUDA10Kernel.h"
#define SALT 3948
#include "CUDA10Kernel.h"
#define SALT 3949
#include "CUDA10Kernel.h"
#define SALT 3950
#include "CUDA10Kernel.h"
#define SALT 3951
#include "CUDA10Kernel.h"
#define SALT 3952
#include "CUDA10Kernel.h"
#define SALT 3953
#include "CUDA10Kernel.h"
#define SALT 3954
#include "CUDA10Kernel.h"
#define SALT 3955
#include "CUDA10Kernel.h"
#define SALT 3956
#include "CUDA10Kernel.h"
#define SALT 3957
#include "CUDA10Kernel.h"
#define SALT 3958
#include "CUDA10Kernel.h"
#define SALT 3959
#include "CUDA10Kernel.h"
#define SALT 3960
#include "CUDA10Kernel.h"
#define SALT 3961
#include "CUDA10Kernel.h"
#define SALT 3962
#include "CUDA10Kernel.h"
#define SALT 3963
#include "CUDA10Kernel.h"
#define SALT 3964
#include "CUDA10Kernel.h"
#define SALT 3965
#include "CUDA10Kernel.h"
#define SALT 3966
#include "CUDA10Kernel.h"
#define SALT 3967
#include "CUDA10Kernel.h"
#define SALT 3968
#include "CUDA10Kernel.h"
#define SALT 3969
#include "CUDA10Kernel.h"
#define SALT 3970
#include "CUDA10Kernel.h"
#define SALT 3971
#include "CUDA10Kernel.h"
#define SALT 3972
#include "CUDA10Kernel.h"
#define SALT 3973
#include "CUDA10Kernel.h"
#define SALT 3974
#include "CUDA10Kernel.h"
#define SALT 3975
#include "CUDA10Kernel.h"
#define SALT 3976
#include "CUDA10Kernel.h"
#define SALT 3977
#include "CUDA10Kernel.h"
#define SALT 3978
#include "CUDA10Kernel.h"
#define SALT 3979
#include "CUDA10Kernel.h"
#define SALT 3980
#include "CUDA10Kernel.h"
#define SALT 3981
#include "CUDA10Kernel.h"
#define SALT 3982
#include "CUDA10Kernel.h"
#define SALT 3983
#include "CUDA10Kernel.h"
#define SALT 3984
#include "CUDA10Kernel.h"
#define SALT 3985
#include "CUDA10Kernel.h"
#define SALT 3986
#include "CUDA10Kernel.h"
#define SALT 3987
#include "CUDA10Kernel.h"
#define SALT 3988
#include "CUDA10Kernel.h"
#define SALT 3989
#include "CUDA10Kernel.h"
#define SALT 3990
#include "CUDA10Kernel.h"
#define SALT 3991
#include "CUDA10Kernel.h"
#define SALT 3992
#include "CUDA10Kernel.h"
#define SALT 3993
#include "CUDA10Kernel.h"
#define SALT 3994
#include "CUDA10Kernel.h"
#define SALT 3995
#include "CUDA10Kernel.h"
#define SALT 3996
#include "CUDA10Kernel.h"
#define SALT 3997
#include "CUDA10Kernel.h"
#define SALT 3998
#include "CUDA10Kernel.h"
#define SALT 3999
#include "CUDA10Kernel.h"
#define SALT 4000
#include "CUDA10Kernel.h"
#define SALT 4001
#include "CUDA10Kernel.h"
#define SALT 4002
#include "CUDA10Kernel.h"
#define SALT 4003
#include "CUDA10Kernel.h"
#define SALT 4004
#include "CUDA10Kernel.h"
#define SALT 4005
#include "CUDA10Kernel.h"
#define SALT 4006
#include "CUDA10Kernel.h"
#define SALT 4007
#include "CUDA10Kernel.h"
#define SALT 4008
#include "CUDA10Kernel.h"
#define SALT 4009
#include "CUDA10Kernel.h"
#define SALT 4010
#include "CUDA10Kernel.h"
#define SALT 4011
#include "CUDA10Kernel.h"
#define SALT 4012
#include "CUDA10Kernel.h"
#define SALT 4013
#include "CUDA10Kernel.h"
#define SALT 4014
#include "CUDA10Kernel.h"
#define SALT 4015
#include "CUDA10Kernel.h"
#define SALT 4016
#include "CUDA10Kernel.h"
#define SALT 4017
#include "CUDA10Kernel.h"
#define SALT 4018
#include "CUDA10Kernel.h"
#define SALT 4019
#include "CUDA10Kernel.h"
#define SALT 4020
#include "CUDA10Kernel.h"
#define SALT 4021
#include "CUDA10Kernel.h"
#define SALT 4022
#include "CUDA10Kernel.h"
#define SALT 4023
#include "CUDA10Kernel.h"
#define SALT 4024
#include "CUDA10Kernel.h"
#define SALT 4025
#include "CUDA10Kernel.h"
#define SALT 4026
#include "CUDA10Kernel.h"
#define SALT 4027
#include "CUDA10Kernel.h"
#define SALT 4028
#include "CUDA10Kernel.h"
#define SALT 4029
#include "CUDA10Kernel.h"
#define SALT 4030
#include "CUDA10Kernel.h"
#define SALT 4031
#include "CUDA10Kernel.h"
#define SALT 4032
#include "CUDA10Kernel.h"
#define SALT 4033
#include "CUDA10Kernel.h"
#define SALT 4034
#include "CUDA10Kernel.h"
#define SALT 4035
#include "CUDA10Kernel.h"
#define SALT 4036
#include "CUDA10Kernel.h"
#define SALT 4037
#include "CUDA10Kernel.h"
#define SALT 4038
#include "CUDA10Kernel.h"
#define SALT 4039
#include "CUDA10Kernel.h"
#define SALT 4040
#include "CUDA10Kernel.h"
#define SALT 4041
#include "CUDA10Kernel.h"
#define SALT 4042
#include "CUDA10Kernel.h"
#define SALT 4043
#include "CUDA10Kernel.h"
#define SALT 4044
#include "CUDA10Kernel.h"
#define SALT 4045
#include "CUDA10Kernel.h"
#define SALT 4046
#include "CUDA10Kernel.h"
#define SALT 4047
#include "CUDA10Kernel.h"
#define SALT 4048
#include "CUDA10Kernel.h"
#define SALT 4049
#include "CUDA10Kernel.h"
#define SALT 4050
#include "CUDA10Kernel.h"
#define SALT 4051
#include "CUDA10Kernel.h"
#define SALT 4052
#include "CUDA10Kernel.h"
#define SALT 4053
#include "CUDA10Kernel.h"
#define SALT 4054
#include "CUDA10Kernel.h"
#define SALT 4055
#include "CUDA10Kernel.h"
#define SALT 4056
#include "CUDA10Kernel.h"
#define SALT 4057
#include "CUDA10Kernel.h"
#define SALT 4058
#include "CUDA10Kernel.h"
#define SALT 4059
#include "CUDA10Kernel.h"
#define SALT 4060
#include "CUDA10Kernel.h"
#define SALT 4061
#include "CUDA10Kernel.h"
#define SALT 4062
#include "CUDA10Kernel.h"
#define SALT 4063
#include "CUDA10Kernel.h"
#define SALT 4064
#include "CUDA10Kernel.h"
#define SALT 4065
#include "CUDA10Kernel.h"
#define SALT 4066
#include "CUDA10Kernel.h"
#define SALT 4067
#include "CUDA10Kernel.h"
#define SALT 4068
#include "CUDA10Kernel.h"
#define SALT 4069
#include "CUDA10Kernel.h"
#define SALT 4070
#include "CUDA10Kernel.h"
#define SALT 4071
#include "CUDA10Kernel.h"
#define SALT 4072
#include "CUDA10Kernel.h"
#define SALT 4073
#include "CUDA10Kernel.h"
#define SALT 4074
#include "CUDA10Kernel.h"
#define SALT 4075
#include "CUDA10Kernel.h"
#define SALT 4076
#include "CUDA10Kernel.h"
#define SALT 4077
#include "CUDA10Kernel.h"
#define SALT 4078
#include "CUDA10Kernel.h"
#define SALT 4079
#include "CUDA10Kernel.h"
#define SALT 4080
#include "CUDA10Kernel.h"
#define SALT 4081
#include "CUDA10Kernel.h"
#define SALT 4082
#include "CUDA10Kernel.h"
#define SALT 4083
#include "CUDA10Kernel.h"
#define SALT 4084
#include "CUDA10Kernel.h"
#define SALT 4085
#include "CUDA10Kernel.h"
#define SALT 4086
#include "CUDA10Kernel.h"
#define SALT 4087
#include "CUDA10Kernel.h"
#define SALT 4088
#include "CUDA10Kernel.h"
#define SALT 4089
#include "CUDA10Kernel.h"
#define SALT 4090
#include "CUDA10Kernel.h"
#define SALT 4091
#include "CUDA10Kernel.h"
#define SALT 4092
#include "CUDA10Kernel.h"
#define SALT 4093
#include "CUDA10Kernel.h"
#define SALT 4094
#include "CUDA10Kernel.h"
#define SALT 4095
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher7()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel7(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 3584: CUDA_DES_PerformSearch3584<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3585: CUDA_DES_PerformSearch3585<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3586: CUDA_DES_PerformSearch3586<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3587: CUDA_DES_PerformSearch3587<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3588: CUDA_DES_PerformSearch3588<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3589: CUDA_DES_PerformSearch3589<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3590: CUDA_DES_PerformSearch3590<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3591: CUDA_DES_PerformSearch3591<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3592: CUDA_DES_PerformSearch3592<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3593: CUDA_DES_PerformSearch3593<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3594: CUDA_DES_PerformSearch3594<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3595: CUDA_DES_PerformSearch3595<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3596: CUDA_DES_PerformSearch3596<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3597: CUDA_DES_PerformSearch3597<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3598: CUDA_DES_PerformSearch3598<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3599: CUDA_DES_PerformSearch3599<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3600: CUDA_DES_PerformSearch3600<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3601: CUDA_DES_PerformSearch3601<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3602: CUDA_DES_PerformSearch3602<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3603: CUDA_DES_PerformSearch3603<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3604: CUDA_DES_PerformSearch3604<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3605: CUDA_DES_PerformSearch3605<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3606: CUDA_DES_PerformSearch3606<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3607: CUDA_DES_PerformSearch3607<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3608: CUDA_DES_PerformSearch3608<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3609: CUDA_DES_PerformSearch3609<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3610: CUDA_DES_PerformSearch3610<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3611: CUDA_DES_PerformSearch3611<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3612: CUDA_DES_PerformSearch3612<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3613: CUDA_DES_PerformSearch3613<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3614: CUDA_DES_PerformSearch3614<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3615: CUDA_DES_PerformSearch3615<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3616: CUDA_DES_PerformSearch3616<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3617: CUDA_DES_PerformSearch3617<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3618: CUDA_DES_PerformSearch3618<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3619: CUDA_DES_PerformSearch3619<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3620: CUDA_DES_PerformSearch3620<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3621: CUDA_DES_PerformSearch3621<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3622: CUDA_DES_PerformSearch3622<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3623: CUDA_DES_PerformSearch3623<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3624: CUDA_DES_PerformSearch3624<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3625: CUDA_DES_PerformSearch3625<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3626: CUDA_DES_PerformSearch3626<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3627: CUDA_DES_PerformSearch3627<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3628: CUDA_DES_PerformSearch3628<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3629: CUDA_DES_PerformSearch3629<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3630: CUDA_DES_PerformSearch3630<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3631: CUDA_DES_PerformSearch3631<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3632: CUDA_DES_PerformSearch3632<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3633: CUDA_DES_PerformSearch3633<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3634: CUDA_DES_PerformSearch3634<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3635: CUDA_DES_PerformSearch3635<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3636: CUDA_DES_PerformSearch3636<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3637: CUDA_DES_PerformSearch3637<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3638: CUDA_DES_PerformSearch3638<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3639: CUDA_DES_PerformSearch3639<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3640: CUDA_DES_PerformSearch3640<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3641: CUDA_DES_PerformSearch3641<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3642: CUDA_DES_PerformSearch3642<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3643: CUDA_DES_PerformSearch3643<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3644: CUDA_DES_PerformSearch3644<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3645: CUDA_DES_PerformSearch3645<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3646: CUDA_DES_PerformSearch3646<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3647: CUDA_DES_PerformSearch3647<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3648: CUDA_DES_PerformSearch3648<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3649: CUDA_DES_PerformSearch3649<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3650: CUDA_DES_PerformSearch3650<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3651: CUDA_DES_PerformSearch3651<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3652: CUDA_DES_PerformSearch3652<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3653: CUDA_DES_PerformSearch3653<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3654: CUDA_DES_PerformSearch3654<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3655: CUDA_DES_PerformSearch3655<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3656: CUDA_DES_PerformSearch3656<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3657: CUDA_DES_PerformSearch3657<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3658: CUDA_DES_PerformSearch3658<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3659: CUDA_DES_PerformSearch3659<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3660: CUDA_DES_PerformSearch3660<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3661: CUDA_DES_PerformSearch3661<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3662: CUDA_DES_PerformSearch3662<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3663: CUDA_DES_PerformSearch3663<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3664: CUDA_DES_PerformSearch3664<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3665: CUDA_DES_PerformSearch3665<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3666: CUDA_DES_PerformSearch3666<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3667: CUDA_DES_PerformSearch3667<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3668: CUDA_DES_PerformSearch3668<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3669: CUDA_DES_PerformSearch3669<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3670: CUDA_DES_PerformSearch3670<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3671: CUDA_DES_PerformSearch3671<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3672: CUDA_DES_PerformSearch3672<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3673: CUDA_DES_PerformSearch3673<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3674: CUDA_DES_PerformSearch3674<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3675: CUDA_DES_PerformSearch3675<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3676: CUDA_DES_PerformSearch3676<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3677: CUDA_DES_PerformSearch3677<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3678: CUDA_DES_PerformSearch3678<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3679: CUDA_DES_PerformSearch3679<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3680: CUDA_DES_PerformSearch3680<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3681: CUDA_DES_PerformSearch3681<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3682: CUDA_DES_PerformSearch3682<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3683: CUDA_DES_PerformSearch3683<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3684: CUDA_DES_PerformSearch3684<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3685: CUDA_DES_PerformSearch3685<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3686: CUDA_DES_PerformSearch3686<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3687: CUDA_DES_PerformSearch3687<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3688: CUDA_DES_PerformSearch3688<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3689: CUDA_DES_PerformSearch3689<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3690: CUDA_DES_PerformSearch3690<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3691: CUDA_DES_PerformSearch3691<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3692: CUDA_DES_PerformSearch3692<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3693: CUDA_DES_PerformSearch3693<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3694: CUDA_DES_PerformSearch3694<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3695: CUDA_DES_PerformSearch3695<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3696: CUDA_DES_PerformSearch3696<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3697: CUDA_DES_PerformSearch3697<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3698: CUDA_DES_PerformSearch3698<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3699: CUDA_DES_PerformSearch3699<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3700: CUDA_DES_PerformSearch3700<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3701: CUDA_DES_PerformSearch3701<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3702: CUDA_DES_PerformSearch3702<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3703: CUDA_DES_PerformSearch3703<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3704: CUDA_DES_PerformSearch3704<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3705: CUDA_DES_PerformSearch3705<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3706: CUDA_DES_PerformSearch3706<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3707: CUDA_DES_PerformSearch3707<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3708: CUDA_DES_PerformSearch3708<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3709: CUDA_DES_PerformSearch3709<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3710: CUDA_DES_PerformSearch3710<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3711: CUDA_DES_PerformSearch3711<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3712: CUDA_DES_PerformSearch3712<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3713: CUDA_DES_PerformSearch3713<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3714: CUDA_DES_PerformSearch3714<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3715: CUDA_DES_PerformSearch3715<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3716: CUDA_DES_PerformSearch3716<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3717: CUDA_DES_PerformSearch3717<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3718: CUDA_DES_PerformSearch3718<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3719: CUDA_DES_PerformSearch3719<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3720: CUDA_DES_PerformSearch3720<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3721: CUDA_DES_PerformSearch3721<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3722: CUDA_DES_PerformSearch3722<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3723: CUDA_DES_PerformSearch3723<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3724: CUDA_DES_PerformSearch3724<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3725: CUDA_DES_PerformSearch3725<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3726: CUDA_DES_PerformSearch3726<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3727: CUDA_DES_PerformSearch3727<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3728: CUDA_DES_PerformSearch3728<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3729: CUDA_DES_PerformSearch3729<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3730: CUDA_DES_PerformSearch3730<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3731: CUDA_DES_PerformSearch3731<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3732: CUDA_DES_PerformSearch3732<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3733: CUDA_DES_PerformSearch3733<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3734: CUDA_DES_PerformSearch3734<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3735: CUDA_DES_PerformSearch3735<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3736: CUDA_DES_PerformSearch3736<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3737: CUDA_DES_PerformSearch3737<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3738: CUDA_DES_PerformSearch3738<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3739: CUDA_DES_PerformSearch3739<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3740: CUDA_DES_PerformSearch3740<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3741: CUDA_DES_PerformSearch3741<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3742: CUDA_DES_PerformSearch3742<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3743: CUDA_DES_PerformSearch3743<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3744: CUDA_DES_PerformSearch3744<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3745: CUDA_DES_PerformSearch3745<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3746: CUDA_DES_PerformSearch3746<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3747: CUDA_DES_PerformSearch3747<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3748: CUDA_DES_PerformSearch3748<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3749: CUDA_DES_PerformSearch3749<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3750: CUDA_DES_PerformSearch3750<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3751: CUDA_DES_PerformSearch3751<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3752: CUDA_DES_PerformSearch3752<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3753: CUDA_DES_PerformSearch3753<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3754: CUDA_DES_PerformSearch3754<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3755: CUDA_DES_PerformSearch3755<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3756: CUDA_DES_PerformSearch3756<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3757: CUDA_DES_PerformSearch3757<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3758: CUDA_DES_PerformSearch3758<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3759: CUDA_DES_PerformSearch3759<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3760: CUDA_DES_PerformSearch3760<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3761: CUDA_DES_PerformSearch3761<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3762: CUDA_DES_PerformSearch3762<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3763: CUDA_DES_PerformSearch3763<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3764: CUDA_DES_PerformSearch3764<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3765: CUDA_DES_PerformSearch3765<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3766: CUDA_DES_PerformSearch3766<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3767: CUDA_DES_PerformSearch3767<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3768: CUDA_DES_PerformSearch3768<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3769: CUDA_DES_PerformSearch3769<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3770: CUDA_DES_PerformSearch3770<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3771: CUDA_DES_PerformSearch3771<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3772: CUDA_DES_PerformSearch3772<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3773: CUDA_DES_PerformSearch3773<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3774: CUDA_DES_PerformSearch3774<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3775: CUDA_DES_PerformSearch3775<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3776: CUDA_DES_PerformSearch3776<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3777: CUDA_DES_PerformSearch3777<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3778: CUDA_DES_PerformSearch3778<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3779: CUDA_DES_PerformSearch3779<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3780: CUDA_DES_PerformSearch3780<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3781: CUDA_DES_PerformSearch3781<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3782: CUDA_DES_PerformSearch3782<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3783: CUDA_DES_PerformSearch3783<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3784: CUDA_DES_PerformSearch3784<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3785: CUDA_DES_PerformSearch3785<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3786: CUDA_DES_PerformSearch3786<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3787: CUDA_DES_PerformSearch3787<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3788: CUDA_DES_PerformSearch3788<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3789: CUDA_DES_PerformSearch3789<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3790: CUDA_DES_PerformSearch3790<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3791: CUDA_DES_PerformSearch3791<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3792: CUDA_DES_PerformSearch3792<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3793: CUDA_DES_PerformSearch3793<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3794: CUDA_DES_PerformSearch3794<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3795: CUDA_DES_PerformSearch3795<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3796: CUDA_DES_PerformSearch3796<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3797: CUDA_DES_PerformSearch3797<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3798: CUDA_DES_PerformSearch3798<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3799: CUDA_DES_PerformSearch3799<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3800: CUDA_DES_PerformSearch3800<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3801: CUDA_DES_PerformSearch3801<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3802: CUDA_DES_PerformSearch3802<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3803: CUDA_DES_PerformSearch3803<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3804: CUDA_DES_PerformSearch3804<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3805: CUDA_DES_PerformSearch3805<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3806: CUDA_DES_PerformSearch3806<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3807: CUDA_DES_PerformSearch3807<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3808: CUDA_DES_PerformSearch3808<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3809: CUDA_DES_PerformSearch3809<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3810: CUDA_DES_PerformSearch3810<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3811: CUDA_DES_PerformSearch3811<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3812: CUDA_DES_PerformSearch3812<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3813: CUDA_DES_PerformSearch3813<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3814: CUDA_DES_PerformSearch3814<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3815: CUDA_DES_PerformSearch3815<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3816: CUDA_DES_PerformSearch3816<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3817: CUDA_DES_PerformSearch3817<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3818: CUDA_DES_PerformSearch3818<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3819: CUDA_DES_PerformSearch3819<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3820: CUDA_DES_PerformSearch3820<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3821: CUDA_DES_PerformSearch3821<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3822: CUDA_DES_PerformSearch3822<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3823: CUDA_DES_PerformSearch3823<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3824: CUDA_DES_PerformSearch3824<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3825: CUDA_DES_PerformSearch3825<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3826: CUDA_DES_PerformSearch3826<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3827: CUDA_DES_PerformSearch3827<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3828: CUDA_DES_PerformSearch3828<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3829: CUDA_DES_PerformSearch3829<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3830: CUDA_DES_PerformSearch3830<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3831: CUDA_DES_PerformSearch3831<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3832: CUDA_DES_PerformSearch3832<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3833: CUDA_DES_PerformSearch3833<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3834: CUDA_DES_PerformSearch3834<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3835: CUDA_DES_PerformSearch3835<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3836: CUDA_DES_PerformSearch3836<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3837: CUDA_DES_PerformSearch3837<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3838: CUDA_DES_PerformSearch3838<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3839: CUDA_DES_PerformSearch3839<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3840: CUDA_DES_PerformSearch3840<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3841: CUDA_DES_PerformSearch3841<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3842: CUDA_DES_PerformSearch3842<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3843: CUDA_DES_PerformSearch3843<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3844: CUDA_DES_PerformSearch3844<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3845: CUDA_DES_PerformSearch3845<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3846: CUDA_DES_PerformSearch3846<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3847: CUDA_DES_PerformSearch3847<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3848: CUDA_DES_PerformSearch3848<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3849: CUDA_DES_PerformSearch3849<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3850: CUDA_DES_PerformSearch3850<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3851: CUDA_DES_PerformSearch3851<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3852: CUDA_DES_PerformSearch3852<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3853: CUDA_DES_PerformSearch3853<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3854: CUDA_DES_PerformSearch3854<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3855: CUDA_DES_PerformSearch3855<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3856: CUDA_DES_PerformSearch3856<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3857: CUDA_DES_PerformSearch3857<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3858: CUDA_DES_PerformSearch3858<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3859: CUDA_DES_PerformSearch3859<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3860: CUDA_DES_PerformSearch3860<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3861: CUDA_DES_PerformSearch3861<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3862: CUDA_DES_PerformSearch3862<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3863: CUDA_DES_PerformSearch3863<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3864: CUDA_DES_PerformSearch3864<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3865: CUDA_DES_PerformSearch3865<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3866: CUDA_DES_PerformSearch3866<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3867: CUDA_DES_PerformSearch3867<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3868: CUDA_DES_PerformSearch3868<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3869: CUDA_DES_PerformSearch3869<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3870: CUDA_DES_PerformSearch3870<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3871: CUDA_DES_PerformSearch3871<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3872: CUDA_DES_PerformSearch3872<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3873: CUDA_DES_PerformSearch3873<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3874: CUDA_DES_PerformSearch3874<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3875: CUDA_DES_PerformSearch3875<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3876: CUDA_DES_PerformSearch3876<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3877: CUDA_DES_PerformSearch3877<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3878: CUDA_DES_PerformSearch3878<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3879: CUDA_DES_PerformSearch3879<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3880: CUDA_DES_PerformSearch3880<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3881: CUDA_DES_PerformSearch3881<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3882: CUDA_DES_PerformSearch3882<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3883: CUDA_DES_PerformSearch3883<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3884: CUDA_DES_PerformSearch3884<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3885: CUDA_DES_PerformSearch3885<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3886: CUDA_DES_PerformSearch3886<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3887: CUDA_DES_PerformSearch3887<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3888: CUDA_DES_PerformSearch3888<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3889: CUDA_DES_PerformSearch3889<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3890: CUDA_DES_PerformSearch3890<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3891: CUDA_DES_PerformSearch3891<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3892: CUDA_DES_PerformSearch3892<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3893: CUDA_DES_PerformSearch3893<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3894: CUDA_DES_PerformSearch3894<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3895: CUDA_DES_PerformSearch3895<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3896: CUDA_DES_PerformSearch3896<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3897: CUDA_DES_PerformSearch3897<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3898: CUDA_DES_PerformSearch3898<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3899: CUDA_DES_PerformSearch3899<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3900: CUDA_DES_PerformSearch3900<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3901: CUDA_DES_PerformSearch3901<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3902: CUDA_DES_PerformSearch3902<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3903: CUDA_DES_PerformSearch3903<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3904: CUDA_DES_PerformSearch3904<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3905: CUDA_DES_PerformSearch3905<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3906: CUDA_DES_PerformSearch3906<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3907: CUDA_DES_PerformSearch3907<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3908: CUDA_DES_PerformSearch3908<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3909: CUDA_DES_PerformSearch3909<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3910: CUDA_DES_PerformSearch3910<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3911: CUDA_DES_PerformSearch3911<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3912: CUDA_DES_PerformSearch3912<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3913: CUDA_DES_PerformSearch3913<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3914: CUDA_DES_PerformSearch3914<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3915: CUDA_DES_PerformSearch3915<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3916: CUDA_DES_PerformSearch3916<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3917: CUDA_DES_PerformSearch3917<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3918: CUDA_DES_PerformSearch3918<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3919: CUDA_DES_PerformSearch3919<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3920: CUDA_DES_PerformSearch3920<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3921: CUDA_DES_PerformSearch3921<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3922: CUDA_DES_PerformSearch3922<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3923: CUDA_DES_PerformSearch3923<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3924: CUDA_DES_PerformSearch3924<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3925: CUDA_DES_PerformSearch3925<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3926: CUDA_DES_PerformSearch3926<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3927: CUDA_DES_PerformSearch3927<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3928: CUDA_DES_PerformSearch3928<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3929: CUDA_DES_PerformSearch3929<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3930: CUDA_DES_PerformSearch3930<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3931: CUDA_DES_PerformSearch3931<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3932: CUDA_DES_PerformSearch3932<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3933: CUDA_DES_PerformSearch3933<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3934: CUDA_DES_PerformSearch3934<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3935: CUDA_DES_PerformSearch3935<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3936: CUDA_DES_PerformSearch3936<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3937: CUDA_DES_PerformSearch3937<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3938: CUDA_DES_PerformSearch3938<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3939: CUDA_DES_PerformSearch3939<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3940: CUDA_DES_PerformSearch3940<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3941: CUDA_DES_PerformSearch3941<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3942: CUDA_DES_PerformSearch3942<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3943: CUDA_DES_PerformSearch3943<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3944: CUDA_DES_PerformSearch3944<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3945: CUDA_DES_PerformSearch3945<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3946: CUDA_DES_PerformSearch3946<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3947: CUDA_DES_PerformSearch3947<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3948: CUDA_DES_PerformSearch3948<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3949: CUDA_DES_PerformSearch3949<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3950: CUDA_DES_PerformSearch3950<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3951: CUDA_DES_PerformSearch3951<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3952: CUDA_DES_PerformSearch3952<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3953: CUDA_DES_PerformSearch3953<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3954: CUDA_DES_PerformSearch3954<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3955: CUDA_DES_PerformSearch3955<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3956: CUDA_DES_PerformSearch3956<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3957: CUDA_DES_PerformSearch3957<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3958: CUDA_DES_PerformSearch3958<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3959: CUDA_DES_PerformSearch3959<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3960: CUDA_DES_PerformSearch3960<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3961: CUDA_DES_PerformSearch3961<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3962: CUDA_DES_PerformSearch3962<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3963: CUDA_DES_PerformSearch3963<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3964: CUDA_DES_PerformSearch3964<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3965: CUDA_DES_PerformSearch3965<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3966: CUDA_DES_PerformSearch3966<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3967: CUDA_DES_PerformSearch3967<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3968: CUDA_DES_PerformSearch3968<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3969: CUDA_DES_PerformSearch3969<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3970: CUDA_DES_PerformSearch3970<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3971: CUDA_DES_PerformSearch3971<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3972: CUDA_DES_PerformSearch3972<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3973: CUDA_DES_PerformSearch3973<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3974: CUDA_DES_PerformSearch3974<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3975: CUDA_DES_PerformSearch3975<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3976: CUDA_DES_PerformSearch3976<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3977: CUDA_DES_PerformSearch3977<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3978: CUDA_DES_PerformSearch3978<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3979: CUDA_DES_PerformSearch3979<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3980: CUDA_DES_PerformSearch3980<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3981: CUDA_DES_PerformSearch3981<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3982: CUDA_DES_PerformSearch3982<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3983: CUDA_DES_PerformSearch3983<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3984: CUDA_DES_PerformSearch3984<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3985: CUDA_DES_PerformSearch3985<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3986: CUDA_DES_PerformSearch3986<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3987: CUDA_DES_PerformSearch3987<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3988: CUDA_DES_PerformSearch3988<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3989: CUDA_DES_PerformSearch3989<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3990: CUDA_DES_PerformSearch3990<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3991: CUDA_DES_PerformSearch3991<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3992: CUDA_DES_PerformSearch3992<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3993: CUDA_DES_PerformSearch3993<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3994: CUDA_DES_PerformSearch3994<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3995: CUDA_DES_PerformSearch3995<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3996: CUDA_DES_PerformSearch3996<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3997: CUDA_DES_PerformSearch3997<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3998: CUDA_DES_PerformSearch3998<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3999: CUDA_DES_PerformSearch3999<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4000: CUDA_DES_PerformSearch4000<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4001: CUDA_DES_PerformSearch4001<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4002: CUDA_DES_PerformSearch4002<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4003: CUDA_DES_PerformSearch4003<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4004: CUDA_DES_PerformSearch4004<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4005: CUDA_DES_PerformSearch4005<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4006: CUDA_DES_PerformSearch4006<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4007: CUDA_DES_PerformSearch4007<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4008: CUDA_DES_PerformSearch4008<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4009: CUDA_DES_PerformSearch4009<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4010: CUDA_DES_PerformSearch4010<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4011: CUDA_DES_PerformSearch4011<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4012: CUDA_DES_PerformSearch4012<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4013: CUDA_DES_PerformSearch4013<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4014: CUDA_DES_PerformSearch4014<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4015: CUDA_DES_PerformSearch4015<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4016: CUDA_DES_PerformSearch4016<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4017: CUDA_DES_PerformSearch4017<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4018: CUDA_DES_PerformSearch4018<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4019: CUDA_DES_PerformSearch4019<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4020: CUDA_DES_PerformSearch4020<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4021: CUDA_DES_PerformSearch4021<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4022: CUDA_DES_PerformSearch4022<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4023: CUDA_DES_PerformSearch4023<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4024: CUDA_DES_PerformSearch4024<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4025: CUDA_DES_PerformSearch4025<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4026: CUDA_DES_PerformSearch4026<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4027: CUDA_DES_PerformSearch4027<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4028: CUDA_DES_PerformSearch4028<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4029: CUDA_DES_PerformSearch4029<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4030: CUDA_DES_PerformSearch4030<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4031: CUDA_DES_PerformSearch4031<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4032: CUDA_DES_PerformSearch4032<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4033: CUDA_DES_PerformSearch4033<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4034: CUDA_DES_PerformSearch4034<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4035: CUDA_DES_PerformSearch4035<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4036: CUDA_DES_PerformSearch4036<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4037: CUDA_DES_PerformSearch4037<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4038: CUDA_DES_PerformSearch4038<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4039: CUDA_DES_PerformSearch4039<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4040: CUDA_DES_PerformSearch4040<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4041: CUDA_DES_PerformSearch4041<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4042: CUDA_DES_PerformSearch4042<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4043: CUDA_DES_PerformSearch4043<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4044: CUDA_DES_PerformSearch4044<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4045: CUDA_DES_PerformSearch4045<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4046: CUDA_DES_PerformSearch4046<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4047: CUDA_DES_PerformSearch4047<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4048: CUDA_DES_PerformSearch4048<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4049: CUDA_DES_PerformSearch4049<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4050: CUDA_DES_PerformSearch4050<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4051: CUDA_DES_PerformSearch4051<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4052: CUDA_DES_PerformSearch4052<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4053: CUDA_DES_PerformSearch4053<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4054: CUDA_DES_PerformSearch4054<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4055: CUDA_DES_PerformSearch4055<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4056: CUDA_DES_PerformSearch4056<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4057: CUDA_DES_PerformSearch4057<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4058: CUDA_DES_PerformSearch4058<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4059: CUDA_DES_PerformSearch4059<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4060: CUDA_DES_PerformSearch4060<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4061: CUDA_DES_PerformSearch4061<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4062: CUDA_DES_PerformSearch4062<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4063: CUDA_DES_PerformSearch4063<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4064: CUDA_DES_PerformSearch4064<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4065: CUDA_DES_PerformSearch4065<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4066: CUDA_DES_PerformSearch4066<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4067: CUDA_DES_PerformSearch4067<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4068: CUDA_DES_PerformSearch4068<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4069: CUDA_DES_PerformSearch4069<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4070: CUDA_DES_PerformSearch4070<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4071: CUDA_DES_PerformSearch4071<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4072: CUDA_DES_PerformSearch4072<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4073: CUDA_DES_PerformSearch4073<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4074: CUDA_DES_PerformSearch4074<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4075: CUDA_DES_PerformSearch4075<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4076: CUDA_DES_PerformSearch4076<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4077: CUDA_DES_PerformSearch4077<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4078: CUDA_DES_PerformSearch4078<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4079: CUDA_DES_PerformSearch4079<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4080: CUDA_DES_PerformSearch4080<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4081: CUDA_DES_PerformSearch4081<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4082: CUDA_DES_PerformSearch4082<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4083: CUDA_DES_PerformSearch4083<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4084: CUDA_DES_PerformSearch4084<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4085: CUDA_DES_PerformSearch4085<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4086: CUDA_DES_PerformSearch4086<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4087: CUDA_DES_PerformSearch4087<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4088: CUDA_DES_PerformSearch4088<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4089: CUDA_DES_PerformSearch4089<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4090: CUDA_DES_PerformSearch4090<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4091: CUDA_DES_PerformSearch4091<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4092: CUDA_DES_PerformSearch4092<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4093: CUDA_DES_PerformSearch4093<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4094: CUDA_DES_PerformSearch4094<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 4095: CUDA_DES_PerformSearch4095<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
