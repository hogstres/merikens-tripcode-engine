// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 1024
#include "CUDA10Kernel.h"
#define SALT 1025
#include "CUDA10Kernel.h"
#define SALT 1026
#include "CUDA10Kernel.h"
#define SALT 1027
#include "CUDA10Kernel.h"
#define SALT 1028
#include "CUDA10Kernel.h"
#define SALT 1029
#include "CUDA10Kernel.h"
#define SALT 1030
#include "CUDA10Kernel.h"
#define SALT 1031
#include "CUDA10Kernel.h"
#define SALT 1032
#include "CUDA10Kernel.h"
#define SALT 1033
#include "CUDA10Kernel.h"
#define SALT 1034
#include "CUDA10Kernel.h"
#define SALT 1035
#include "CUDA10Kernel.h"
#define SALT 1036
#include "CUDA10Kernel.h"
#define SALT 1037
#include "CUDA10Kernel.h"
#define SALT 1038
#include "CUDA10Kernel.h"
#define SALT 1039
#include "CUDA10Kernel.h"
#define SALT 1040
#include "CUDA10Kernel.h"
#define SALT 1041
#include "CUDA10Kernel.h"
#define SALT 1042
#include "CUDA10Kernel.h"
#define SALT 1043
#include "CUDA10Kernel.h"
#define SALT 1044
#include "CUDA10Kernel.h"
#define SALT 1045
#include "CUDA10Kernel.h"
#define SALT 1046
#include "CUDA10Kernel.h"
#define SALT 1047
#include "CUDA10Kernel.h"
#define SALT 1048
#include "CUDA10Kernel.h"
#define SALT 1049
#include "CUDA10Kernel.h"
#define SALT 1050
#include "CUDA10Kernel.h"
#define SALT 1051
#include "CUDA10Kernel.h"
#define SALT 1052
#include "CUDA10Kernel.h"
#define SALT 1053
#include "CUDA10Kernel.h"
#define SALT 1054
#include "CUDA10Kernel.h"
#define SALT 1055
#include "CUDA10Kernel.h"
#define SALT 1056
#include "CUDA10Kernel.h"
#define SALT 1057
#include "CUDA10Kernel.h"
#define SALT 1058
#include "CUDA10Kernel.h"
#define SALT 1059
#include "CUDA10Kernel.h"
#define SALT 1060
#include "CUDA10Kernel.h"
#define SALT 1061
#include "CUDA10Kernel.h"
#define SALT 1062
#include "CUDA10Kernel.h"
#define SALT 1063
#include "CUDA10Kernel.h"
#define SALT 1064
#include "CUDA10Kernel.h"
#define SALT 1065
#include "CUDA10Kernel.h"
#define SALT 1066
#include "CUDA10Kernel.h"
#define SALT 1067
#include "CUDA10Kernel.h"
#define SALT 1068
#include "CUDA10Kernel.h"
#define SALT 1069
#include "CUDA10Kernel.h"
#define SALT 1070
#include "CUDA10Kernel.h"
#define SALT 1071
#include "CUDA10Kernel.h"
#define SALT 1072
#include "CUDA10Kernel.h"
#define SALT 1073
#include "CUDA10Kernel.h"
#define SALT 1074
#include "CUDA10Kernel.h"
#define SALT 1075
#include "CUDA10Kernel.h"
#define SALT 1076
#include "CUDA10Kernel.h"
#define SALT 1077
#include "CUDA10Kernel.h"
#define SALT 1078
#include "CUDA10Kernel.h"
#define SALT 1079
#include "CUDA10Kernel.h"
#define SALT 1080
#include "CUDA10Kernel.h"
#define SALT 1081
#include "CUDA10Kernel.h"
#define SALT 1082
#include "CUDA10Kernel.h"
#define SALT 1083
#include "CUDA10Kernel.h"
#define SALT 1084
#include "CUDA10Kernel.h"
#define SALT 1085
#include "CUDA10Kernel.h"
#define SALT 1086
#include "CUDA10Kernel.h"
#define SALT 1087
#include "CUDA10Kernel.h"
#define SALT 1088
#include "CUDA10Kernel.h"
#define SALT 1089
#include "CUDA10Kernel.h"
#define SALT 1090
#include "CUDA10Kernel.h"
#define SALT 1091
#include "CUDA10Kernel.h"
#define SALT 1092
#include "CUDA10Kernel.h"
#define SALT 1093
#include "CUDA10Kernel.h"
#define SALT 1094
#include "CUDA10Kernel.h"
#define SALT 1095
#include "CUDA10Kernel.h"
#define SALT 1096
#include "CUDA10Kernel.h"
#define SALT 1097
#include "CUDA10Kernel.h"
#define SALT 1098
#include "CUDA10Kernel.h"
#define SALT 1099
#include "CUDA10Kernel.h"
#define SALT 1100
#include "CUDA10Kernel.h"
#define SALT 1101
#include "CUDA10Kernel.h"
#define SALT 1102
#include "CUDA10Kernel.h"
#define SALT 1103
#include "CUDA10Kernel.h"
#define SALT 1104
#include "CUDA10Kernel.h"
#define SALT 1105
#include "CUDA10Kernel.h"
#define SALT 1106
#include "CUDA10Kernel.h"
#define SALT 1107
#include "CUDA10Kernel.h"
#define SALT 1108
#include "CUDA10Kernel.h"
#define SALT 1109
#include "CUDA10Kernel.h"
#define SALT 1110
#include "CUDA10Kernel.h"
#define SALT 1111
#include "CUDA10Kernel.h"
#define SALT 1112
#include "CUDA10Kernel.h"
#define SALT 1113
#include "CUDA10Kernel.h"
#define SALT 1114
#include "CUDA10Kernel.h"
#define SALT 1115
#include "CUDA10Kernel.h"
#define SALT 1116
#include "CUDA10Kernel.h"
#define SALT 1117
#include "CUDA10Kernel.h"
#define SALT 1118
#include "CUDA10Kernel.h"
#define SALT 1119
#include "CUDA10Kernel.h"
#define SALT 1120
#include "CUDA10Kernel.h"
#define SALT 1121
#include "CUDA10Kernel.h"
#define SALT 1122
#include "CUDA10Kernel.h"
#define SALT 1123
#include "CUDA10Kernel.h"
#define SALT 1124
#include "CUDA10Kernel.h"
#define SALT 1125
#include "CUDA10Kernel.h"
#define SALT 1126
#include "CUDA10Kernel.h"
#define SALT 1127
#include "CUDA10Kernel.h"
#define SALT 1128
#include "CUDA10Kernel.h"
#define SALT 1129
#include "CUDA10Kernel.h"
#define SALT 1130
#include "CUDA10Kernel.h"
#define SALT 1131
#include "CUDA10Kernel.h"
#define SALT 1132
#include "CUDA10Kernel.h"
#define SALT 1133
#include "CUDA10Kernel.h"
#define SALT 1134
#include "CUDA10Kernel.h"
#define SALT 1135
#include "CUDA10Kernel.h"
#define SALT 1136
#include "CUDA10Kernel.h"
#define SALT 1137
#include "CUDA10Kernel.h"
#define SALT 1138
#include "CUDA10Kernel.h"
#define SALT 1139
#include "CUDA10Kernel.h"
#define SALT 1140
#include "CUDA10Kernel.h"
#define SALT 1141
#include "CUDA10Kernel.h"
#define SALT 1142
#include "CUDA10Kernel.h"
#define SALT 1143
#include "CUDA10Kernel.h"
#define SALT 1144
#include "CUDA10Kernel.h"
#define SALT 1145
#include "CUDA10Kernel.h"
#define SALT 1146
#include "CUDA10Kernel.h"
#define SALT 1147
#include "CUDA10Kernel.h"
#define SALT 1148
#include "CUDA10Kernel.h"
#define SALT 1149
#include "CUDA10Kernel.h"
#define SALT 1150
#include "CUDA10Kernel.h"
#define SALT 1151
#include "CUDA10Kernel.h"
#define SALT 1152
#include "CUDA10Kernel.h"
#define SALT 1153
#include "CUDA10Kernel.h"
#define SALT 1154
#include "CUDA10Kernel.h"
#define SALT 1155
#include "CUDA10Kernel.h"
#define SALT 1156
#include "CUDA10Kernel.h"
#define SALT 1157
#include "CUDA10Kernel.h"
#define SALT 1158
#include "CUDA10Kernel.h"
#define SALT 1159
#include "CUDA10Kernel.h"
#define SALT 1160
#include "CUDA10Kernel.h"
#define SALT 1161
#include "CUDA10Kernel.h"
#define SALT 1162
#include "CUDA10Kernel.h"
#define SALT 1163
#include "CUDA10Kernel.h"
#define SALT 1164
#include "CUDA10Kernel.h"
#define SALT 1165
#include "CUDA10Kernel.h"
#define SALT 1166
#include "CUDA10Kernel.h"
#define SALT 1167
#include "CUDA10Kernel.h"
#define SALT 1168
#include "CUDA10Kernel.h"
#define SALT 1169
#include "CUDA10Kernel.h"
#define SALT 1170
#include "CUDA10Kernel.h"
#define SALT 1171
#include "CUDA10Kernel.h"
#define SALT 1172
#include "CUDA10Kernel.h"
#define SALT 1173
#include "CUDA10Kernel.h"
#define SALT 1174
#include "CUDA10Kernel.h"
#define SALT 1175
#include "CUDA10Kernel.h"
#define SALT 1176
#include "CUDA10Kernel.h"
#define SALT 1177
#include "CUDA10Kernel.h"
#define SALT 1178
#include "CUDA10Kernel.h"
#define SALT 1179
#include "CUDA10Kernel.h"
#define SALT 1180
#include "CUDA10Kernel.h"
#define SALT 1181
#include "CUDA10Kernel.h"
#define SALT 1182
#include "CUDA10Kernel.h"
#define SALT 1183
#include "CUDA10Kernel.h"
#define SALT 1184
#include "CUDA10Kernel.h"
#define SALT 1185
#include "CUDA10Kernel.h"
#define SALT 1186
#include "CUDA10Kernel.h"
#define SALT 1187
#include "CUDA10Kernel.h"
#define SALT 1188
#include "CUDA10Kernel.h"
#define SALT 1189
#include "CUDA10Kernel.h"
#define SALT 1190
#include "CUDA10Kernel.h"
#define SALT 1191
#include "CUDA10Kernel.h"
#define SALT 1192
#include "CUDA10Kernel.h"
#define SALT 1193
#include "CUDA10Kernel.h"
#define SALT 1194
#include "CUDA10Kernel.h"
#define SALT 1195
#include "CUDA10Kernel.h"
#define SALT 1196
#include "CUDA10Kernel.h"
#define SALT 1197
#include "CUDA10Kernel.h"
#define SALT 1198
#include "CUDA10Kernel.h"
#define SALT 1199
#include "CUDA10Kernel.h"
#define SALT 1200
#include "CUDA10Kernel.h"
#define SALT 1201
#include "CUDA10Kernel.h"
#define SALT 1202
#include "CUDA10Kernel.h"
#define SALT 1203
#include "CUDA10Kernel.h"
#define SALT 1204
#include "CUDA10Kernel.h"
#define SALT 1205
#include "CUDA10Kernel.h"
#define SALT 1206
#include "CUDA10Kernel.h"
#define SALT 1207
#include "CUDA10Kernel.h"
#define SALT 1208
#include "CUDA10Kernel.h"
#define SALT 1209
#include "CUDA10Kernel.h"
#define SALT 1210
#include "CUDA10Kernel.h"
#define SALT 1211
#include "CUDA10Kernel.h"
#define SALT 1212
#include "CUDA10Kernel.h"
#define SALT 1213
#include "CUDA10Kernel.h"
#define SALT 1214
#include "CUDA10Kernel.h"
#define SALT 1215
#include "CUDA10Kernel.h"
#define SALT 1216
#include "CUDA10Kernel.h"
#define SALT 1217
#include "CUDA10Kernel.h"
#define SALT 1218
#include "CUDA10Kernel.h"
#define SALT 1219
#include "CUDA10Kernel.h"
#define SALT 1220
#include "CUDA10Kernel.h"
#define SALT 1221
#include "CUDA10Kernel.h"
#define SALT 1222
#include "CUDA10Kernel.h"
#define SALT 1223
#include "CUDA10Kernel.h"
#define SALT 1224
#include "CUDA10Kernel.h"
#define SALT 1225
#include "CUDA10Kernel.h"
#define SALT 1226
#include "CUDA10Kernel.h"
#define SALT 1227
#include "CUDA10Kernel.h"
#define SALT 1228
#include "CUDA10Kernel.h"
#define SALT 1229
#include "CUDA10Kernel.h"
#define SALT 1230
#include "CUDA10Kernel.h"
#define SALT 1231
#include "CUDA10Kernel.h"
#define SALT 1232
#include "CUDA10Kernel.h"
#define SALT 1233
#include "CUDA10Kernel.h"
#define SALT 1234
#include "CUDA10Kernel.h"
#define SALT 1235
#include "CUDA10Kernel.h"
#define SALT 1236
#include "CUDA10Kernel.h"
#define SALT 1237
#include "CUDA10Kernel.h"
#define SALT 1238
#include "CUDA10Kernel.h"
#define SALT 1239
#include "CUDA10Kernel.h"
#define SALT 1240
#include "CUDA10Kernel.h"
#define SALT 1241
#include "CUDA10Kernel.h"
#define SALT 1242
#include "CUDA10Kernel.h"
#define SALT 1243
#include "CUDA10Kernel.h"
#define SALT 1244
#include "CUDA10Kernel.h"
#define SALT 1245
#include "CUDA10Kernel.h"
#define SALT 1246
#include "CUDA10Kernel.h"
#define SALT 1247
#include "CUDA10Kernel.h"
#define SALT 1248
#include "CUDA10Kernel.h"
#define SALT 1249
#include "CUDA10Kernel.h"
#define SALT 1250
#include "CUDA10Kernel.h"
#define SALT 1251
#include "CUDA10Kernel.h"
#define SALT 1252
#include "CUDA10Kernel.h"
#define SALT 1253
#include "CUDA10Kernel.h"
#define SALT 1254
#include "CUDA10Kernel.h"
#define SALT 1255
#include "CUDA10Kernel.h"
#define SALT 1256
#include "CUDA10Kernel.h"
#define SALT 1257
#include "CUDA10Kernel.h"
#define SALT 1258
#include "CUDA10Kernel.h"
#define SALT 1259
#include "CUDA10Kernel.h"
#define SALT 1260
#include "CUDA10Kernel.h"
#define SALT 1261
#include "CUDA10Kernel.h"
#define SALT 1262
#include "CUDA10Kernel.h"
#define SALT 1263
#include "CUDA10Kernel.h"
#define SALT 1264
#include "CUDA10Kernel.h"
#define SALT 1265
#include "CUDA10Kernel.h"
#define SALT 1266
#include "CUDA10Kernel.h"
#define SALT 1267
#include "CUDA10Kernel.h"
#define SALT 1268
#include "CUDA10Kernel.h"
#define SALT 1269
#include "CUDA10Kernel.h"
#define SALT 1270
#include "CUDA10Kernel.h"
#define SALT 1271
#include "CUDA10Kernel.h"
#define SALT 1272
#include "CUDA10Kernel.h"
#define SALT 1273
#include "CUDA10Kernel.h"
#define SALT 1274
#include "CUDA10Kernel.h"
#define SALT 1275
#include "CUDA10Kernel.h"
#define SALT 1276
#include "CUDA10Kernel.h"
#define SALT 1277
#include "CUDA10Kernel.h"
#define SALT 1278
#include "CUDA10Kernel.h"
#define SALT 1279
#include "CUDA10Kernel.h"
#define SALT 1280
#include "CUDA10Kernel.h"
#define SALT 1281
#include "CUDA10Kernel.h"
#define SALT 1282
#include "CUDA10Kernel.h"
#define SALT 1283
#include "CUDA10Kernel.h"
#define SALT 1284
#include "CUDA10Kernel.h"
#define SALT 1285
#include "CUDA10Kernel.h"
#define SALT 1286
#include "CUDA10Kernel.h"
#define SALT 1287
#include "CUDA10Kernel.h"
#define SALT 1288
#include "CUDA10Kernel.h"
#define SALT 1289
#include "CUDA10Kernel.h"
#define SALT 1290
#include "CUDA10Kernel.h"
#define SALT 1291
#include "CUDA10Kernel.h"
#define SALT 1292
#include "CUDA10Kernel.h"
#define SALT 1293
#include "CUDA10Kernel.h"
#define SALT 1294
#include "CUDA10Kernel.h"
#define SALT 1295
#include "CUDA10Kernel.h"
#define SALT 1296
#include "CUDA10Kernel.h"
#define SALT 1297
#include "CUDA10Kernel.h"
#define SALT 1298
#include "CUDA10Kernel.h"
#define SALT 1299
#include "CUDA10Kernel.h"
#define SALT 1300
#include "CUDA10Kernel.h"
#define SALT 1301
#include "CUDA10Kernel.h"
#define SALT 1302
#include "CUDA10Kernel.h"
#define SALT 1303
#include "CUDA10Kernel.h"
#define SALT 1304
#include "CUDA10Kernel.h"
#define SALT 1305
#include "CUDA10Kernel.h"
#define SALT 1306
#include "CUDA10Kernel.h"
#define SALT 1307
#include "CUDA10Kernel.h"
#define SALT 1308
#include "CUDA10Kernel.h"
#define SALT 1309
#include "CUDA10Kernel.h"
#define SALT 1310
#include "CUDA10Kernel.h"
#define SALT 1311
#include "CUDA10Kernel.h"
#define SALT 1312
#include "CUDA10Kernel.h"
#define SALT 1313
#include "CUDA10Kernel.h"
#define SALT 1314
#include "CUDA10Kernel.h"
#define SALT 1315
#include "CUDA10Kernel.h"
#define SALT 1316
#include "CUDA10Kernel.h"
#define SALT 1317
#include "CUDA10Kernel.h"
#define SALT 1318
#include "CUDA10Kernel.h"
#define SALT 1319
#include "CUDA10Kernel.h"
#define SALT 1320
#include "CUDA10Kernel.h"
#define SALT 1321
#include "CUDA10Kernel.h"
#define SALT 1322
#include "CUDA10Kernel.h"
#define SALT 1323
#include "CUDA10Kernel.h"
#define SALT 1324
#include "CUDA10Kernel.h"
#define SALT 1325
#include "CUDA10Kernel.h"
#define SALT 1326
#include "CUDA10Kernel.h"
#define SALT 1327
#include "CUDA10Kernel.h"
#define SALT 1328
#include "CUDA10Kernel.h"
#define SALT 1329
#include "CUDA10Kernel.h"
#define SALT 1330
#include "CUDA10Kernel.h"
#define SALT 1331
#include "CUDA10Kernel.h"
#define SALT 1332
#include "CUDA10Kernel.h"
#define SALT 1333
#include "CUDA10Kernel.h"
#define SALT 1334
#include "CUDA10Kernel.h"
#define SALT 1335
#include "CUDA10Kernel.h"
#define SALT 1336
#include "CUDA10Kernel.h"
#define SALT 1337
#include "CUDA10Kernel.h"
#define SALT 1338
#include "CUDA10Kernel.h"
#define SALT 1339
#include "CUDA10Kernel.h"
#define SALT 1340
#include "CUDA10Kernel.h"
#define SALT 1341
#include "CUDA10Kernel.h"
#define SALT 1342
#include "CUDA10Kernel.h"
#define SALT 1343
#include "CUDA10Kernel.h"
#define SALT 1344
#include "CUDA10Kernel.h"
#define SALT 1345
#include "CUDA10Kernel.h"
#define SALT 1346
#include "CUDA10Kernel.h"
#define SALT 1347
#include "CUDA10Kernel.h"
#define SALT 1348
#include "CUDA10Kernel.h"
#define SALT 1349
#include "CUDA10Kernel.h"
#define SALT 1350
#include "CUDA10Kernel.h"
#define SALT 1351
#include "CUDA10Kernel.h"
#define SALT 1352
#include "CUDA10Kernel.h"
#define SALT 1353
#include "CUDA10Kernel.h"
#define SALT 1354
#include "CUDA10Kernel.h"
#define SALT 1355
#include "CUDA10Kernel.h"
#define SALT 1356
#include "CUDA10Kernel.h"
#define SALT 1357
#include "CUDA10Kernel.h"
#define SALT 1358
#include "CUDA10Kernel.h"
#define SALT 1359
#include "CUDA10Kernel.h"
#define SALT 1360
#include "CUDA10Kernel.h"
#define SALT 1361
#include "CUDA10Kernel.h"
#define SALT 1362
#include "CUDA10Kernel.h"
#define SALT 1363
#include "CUDA10Kernel.h"
#define SALT 1364
#include "CUDA10Kernel.h"
#define SALT 1365
#include "CUDA10Kernel.h"
#define SALT 1366
#include "CUDA10Kernel.h"
#define SALT 1367
#include "CUDA10Kernel.h"
#define SALT 1368
#include "CUDA10Kernel.h"
#define SALT 1369
#include "CUDA10Kernel.h"
#define SALT 1370
#include "CUDA10Kernel.h"
#define SALT 1371
#include "CUDA10Kernel.h"
#define SALT 1372
#include "CUDA10Kernel.h"
#define SALT 1373
#include "CUDA10Kernel.h"
#define SALT 1374
#include "CUDA10Kernel.h"
#define SALT 1375
#include "CUDA10Kernel.h"
#define SALT 1376
#include "CUDA10Kernel.h"
#define SALT 1377
#include "CUDA10Kernel.h"
#define SALT 1378
#include "CUDA10Kernel.h"
#define SALT 1379
#include "CUDA10Kernel.h"
#define SALT 1380
#include "CUDA10Kernel.h"
#define SALT 1381
#include "CUDA10Kernel.h"
#define SALT 1382
#include "CUDA10Kernel.h"
#define SALT 1383
#include "CUDA10Kernel.h"
#define SALT 1384
#include "CUDA10Kernel.h"
#define SALT 1385
#include "CUDA10Kernel.h"
#define SALT 1386
#include "CUDA10Kernel.h"
#define SALT 1387
#include "CUDA10Kernel.h"
#define SALT 1388
#include "CUDA10Kernel.h"
#define SALT 1389
#include "CUDA10Kernel.h"
#define SALT 1390
#include "CUDA10Kernel.h"
#define SALT 1391
#include "CUDA10Kernel.h"
#define SALT 1392
#include "CUDA10Kernel.h"
#define SALT 1393
#include "CUDA10Kernel.h"
#define SALT 1394
#include "CUDA10Kernel.h"
#define SALT 1395
#include "CUDA10Kernel.h"
#define SALT 1396
#include "CUDA10Kernel.h"
#define SALT 1397
#include "CUDA10Kernel.h"
#define SALT 1398
#include "CUDA10Kernel.h"
#define SALT 1399
#include "CUDA10Kernel.h"
#define SALT 1400
#include "CUDA10Kernel.h"
#define SALT 1401
#include "CUDA10Kernel.h"
#define SALT 1402
#include "CUDA10Kernel.h"
#define SALT 1403
#include "CUDA10Kernel.h"
#define SALT 1404
#include "CUDA10Kernel.h"
#define SALT 1405
#include "CUDA10Kernel.h"
#define SALT 1406
#include "CUDA10Kernel.h"
#define SALT 1407
#include "CUDA10Kernel.h"
#define SALT 1408
#include "CUDA10Kernel.h"
#define SALT 1409
#include "CUDA10Kernel.h"
#define SALT 1410
#include "CUDA10Kernel.h"
#define SALT 1411
#include "CUDA10Kernel.h"
#define SALT 1412
#include "CUDA10Kernel.h"
#define SALT 1413
#include "CUDA10Kernel.h"
#define SALT 1414
#include "CUDA10Kernel.h"
#define SALT 1415
#include "CUDA10Kernel.h"
#define SALT 1416
#include "CUDA10Kernel.h"
#define SALT 1417
#include "CUDA10Kernel.h"
#define SALT 1418
#include "CUDA10Kernel.h"
#define SALT 1419
#include "CUDA10Kernel.h"
#define SALT 1420
#include "CUDA10Kernel.h"
#define SALT 1421
#include "CUDA10Kernel.h"
#define SALT 1422
#include "CUDA10Kernel.h"
#define SALT 1423
#include "CUDA10Kernel.h"
#define SALT 1424
#include "CUDA10Kernel.h"
#define SALT 1425
#include "CUDA10Kernel.h"
#define SALT 1426
#include "CUDA10Kernel.h"
#define SALT 1427
#include "CUDA10Kernel.h"
#define SALT 1428
#include "CUDA10Kernel.h"
#define SALT 1429
#include "CUDA10Kernel.h"
#define SALT 1430
#include "CUDA10Kernel.h"
#define SALT 1431
#include "CUDA10Kernel.h"
#define SALT 1432
#include "CUDA10Kernel.h"
#define SALT 1433
#include "CUDA10Kernel.h"
#define SALT 1434
#include "CUDA10Kernel.h"
#define SALT 1435
#include "CUDA10Kernel.h"
#define SALT 1436
#include "CUDA10Kernel.h"
#define SALT 1437
#include "CUDA10Kernel.h"
#define SALT 1438
#include "CUDA10Kernel.h"
#define SALT 1439
#include "CUDA10Kernel.h"
#define SALT 1440
#include "CUDA10Kernel.h"
#define SALT 1441
#include "CUDA10Kernel.h"
#define SALT 1442
#include "CUDA10Kernel.h"
#define SALT 1443
#include "CUDA10Kernel.h"
#define SALT 1444
#include "CUDA10Kernel.h"
#define SALT 1445
#include "CUDA10Kernel.h"
#define SALT 1446
#include "CUDA10Kernel.h"
#define SALT 1447
#include "CUDA10Kernel.h"
#define SALT 1448
#include "CUDA10Kernel.h"
#define SALT 1449
#include "CUDA10Kernel.h"
#define SALT 1450
#include "CUDA10Kernel.h"
#define SALT 1451
#include "CUDA10Kernel.h"
#define SALT 1452
#include "CUDA10Kernel.h"
#define SALT 1453
#include "CUDA10Kernel.h"
#define SALT 1454
#include "CUDA10Kernel.h"
#define SALT 1455
#include "CUDA10Kernel.h"
#define SALT 1456
#include "CUDA10Kernel.h"
#define SALT 1457
#include "CUDA10Kernel.h"
#define SALT 1458
#include "CUDA10Kernel.h"
#define SALT 1459
#include "CUDA10Kernel.h"
#define SALT 1460
#include "CUDA10Kernel.h"
#define SALT 1461
#include "CUDA10Kernel.h"
#define SALT 1462
#include "CUDA10Kernel.h"
#define SALT 1463
#include "CUDA10Kernel.h"
#define SALT 1464
#include "CUDA10Kernel.h"
#define SALT 1465
#include "CUDA10Kernel.h"
#define SALT 1466
#include "CUDA10Kernel.h"
#define SALT 1467
#include "CUDA10Kernel.h"
#define SALT 1468
#include "CUDA10Kernel.h"
#define SALT 1469
#include "CUDA10Kernel.h"
#define SALT 1470
#include "CUDA10Kernel.h"
#define SALT 1471
#include "CUDA10Kernel.h"
#define SALT 1472
#include "CUDA10Kernel.h"
#define SALT 1473
#include "CUDA10Kernel.h"
#define SALT 1474
#include "CUDA10Kernel.h"
#define SALT 1475
#include "CUDA10Kernel.h"
#define SALT 1476
#include "CUDA10Kernel.h"
#define SALT 1477
#include "CUDA10Kernel.h"
#define SALT 1478
#include "CUDA10Kernel.h"
#define SALT 1479
#include "CUDA10Kernel.h"
#define SALT 1480
#include "CUDA10Kernel.h"
#define SALT 1481
#include "CUDA10Kernel.h"
#define SALT 1482
#include "CUDA10Kernel.h"
#define SALT 1483
#include "CUDA10Kernel.h"
#define SALT 1484
#include "CUDA10Kernel.h"
#define SALT 1485
#include "CUDA10Kernel.h"
#define SALT 1486
#include "CUDA10Kernel.h"
#define SALT 1487
#include "CUDA10Kernel.h"
#define SALT 1488
#include "CUDA10Kernel.h"
#define SALT 1489
#include "CUDA10Kernel.h"
#define SALT 1490
#include "CUDA10Kernel.h"
#define SALT 1491
#include "CUDA10Kernel.h"
#define SALT 1492
#include "CUDA10Kernel.h"
#define SALT 1493
#include "CUDA10Kernel.h"
#define SALT 1494
#include "CUDA10Kernel.h"
#define SALT 1495
#include "CUDA10Kernel.h"
#define SALT 1496
#include "CUDA10Kernel.h"
#define SALT 1497
#include "CUDA10Kernel.h"
#define SALT 1498
#include "CUDA10Kernel.h"
#define SALT 1499
#include "CUDA10Kernel.h"
#define SALT 1500
#include "CUDA10Kernel.h"
#define SALT 1501
#include "CUDA10Kernel.h"
#define SALT 1502
#include "CUDA10Kernel.h"
#define SALT 1503
#include "CUDA10Kernel.h"
#define SALT 1504
#include "CUDA10Kernel.h"
#define SALT 1505
#include "CUDA10Kernel.h"
#define SALT 1506
#include "CUDA10Kernel.h"
#define SALT 1507
#include "CUDA10Kernel.h"
#define SALT 1508
#include "CUDA10Kernel.h"
#define SALT 1509
#include "CUDA10Kernel.h"
#define SALT 1510
#include "CUDA10Kernel.h"
#define SALT 1511
#include "CUDA10Kernel.h"
#define SALT 1512
#include "CUDA10Kernel.h"
#define SALT 1513
#include "CUDA10Kernel.h"
#define SALT 1514
#include "CUDA10Kernel.h"
#define SALT 1515
#include "CUDA10Kernel.h"
#define SALT 1516
#include "CUDA10Kernel.h"
#define SALT 1517
#include "CUDA10Kernel.h"
#define SALT 1518
#include "CUDA10Kernel.h"
#define SALT 1519
#include "CUDA10Kernel.h"
#define SALT 1520
#include "CUDA10Kernel.h"
#define SALT 1521
#include "CUDA10Kernel.h"
#define SALT 1522
#include "CUDA10Kernel.h"
#define SALT 1523
#include "CUDA10Kernel.h"
#define SALT 1524
#include "CUDA10Kernel.h"
#define SALT 1525
#include "CUDA10Kernel.h"
#define SALT 1526
#include "CUDA10Kernel.h"
#define SALT 1527
#include "CUDA10Kernel.h"
#define SALT 1528
#include "CUDA10Kernel.h"
#define SALT 1529
#include "CUDA10Kernel.h"
#define SALT 1530
#include "CUDA10Kernel.h"
#define SALT 1531
#include "CUDA10Kernel.h"
#define SALT 1532
#include "CUDA10Kernel.h"
#define SALT 1533
#include "CUDA10Kernel.h"
#define SALT 1534
#include "CUDA10Kernel.h"
#define SALT 1535
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher2()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel2(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 1024: CUDA_DES_PerformSearch1024<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1025: CUDA_DES_PerformSearch1025<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1026: CUDA_DES_PerformSearch1026<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1027: CUDA_DES_PerformSearch1027<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1028: CUDA_DES_PerformSearch1028<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1029: CUDA_DES_PerformSearch1029<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1030: CUDA_DES_PerformSearch1030<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1031: CUDA_DES_PerformSearch1031<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1032: CUDA_DES_PerformSearch1032<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1033: CUDA_DES_PerformSearch1033<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1034: CUDA_DES_PerformSearch1034<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1035: CUDA_DES_PerformSearch1035<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1036: CUDA_DES_PerformSearch1036<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1037: CUDA_DES_PerformSearch1037<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1038: CUDA_DES_PerformSearch1038<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1039: CUDA_DES_PerformSearch1039<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1040: CUDA_DES_PerformSearch1040<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1041: CUDA_DES_PerformSearch1041<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1042: CUDA_DES_PerformSearch1042<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1043: CUDA_DES_PerformSearch1043<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1044: CUDA_DES_PerformSearch1044<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1045: CUDA_DES_PerformSearch1045<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1046: CUDA_DES_PerformSearch1046<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1047: CUDA_DES_PerformSearch1047<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1048: CUDA_DES_PerformSearch1048<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1049: CUDA_DES_PerformSearch1049<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1050: CUDA_DES_PerformSearch1050<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1051: CUDA_DES_PerformSearch1051<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1052: CUDA_DES_PerformSearch1052<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1053: CUDA_DES_PerformSearch1053<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1054: CUDA_DES_PerformSearch1054<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1055: CUDA_DES_PerformSearch1055<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1056: CUDA_DES_PerformSearch1056<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1057: CUDA_DES_PerformSearch1057<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1058: CUDA_DES_PerformSearch1058<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1059: CUDA_DES_PerformSearch1059<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1060: CUDA_DES_PerformSearch1060<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1061: CUDA_DES_PerformSearch1061<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1062: CUDA_DES_PerformSearch1062<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1063: CUDA_DES_PerformSearch1063<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1064: CUDA_DES_PerformSearch1064<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1065: CUDA_DES_PerformSearch1065<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1066: CUDA_DES_PerformSearch1066<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1067: CUDA_DES_PerformSearch1067<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1068: CUDA_DES_PerformSearch1068<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1069: CUDA_DES_PerformSearch1069<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1070: CUDA_DES_PerformSearch1070<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1071: CUDA_DES_PerformSearch1071<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1072: CUDA_DES_PerformSearch1072<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1073: CUDA_DES_PerformSearch1073<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1074: CUDA_DES_PerformSearch1074<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1075: CUDA_DES_PerformSearch1075<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1076: CUDA_DES_PerformSearch1076<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1077: CUDA_DES_PerformSearch1077<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1078: CUDA_DES_PerformSearch1078<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1079: CUDA_DES_PerformSearch1079<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1080: CUDA_DES_PerformSearch1080<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1081: CUDA_DES_PerformSearch1081<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1082: CUDA_DES_PerformSearch1082<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1083: CUDA_DES_PerformSearch1083<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1084: CUDA_DES_PerformSearch1084<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1085: CUDA_DES_PerformSearch1085<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1086: CUDA_DES_PerformSearch1086<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1087: CUDA_DES_PerformSearch1087<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1088: CUDA_DES_PerformSearch1088<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1089: CUDA_DES_PerformSearch1089<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1090: CUDA_DES_PerformSearch1090<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1091: CUDA_DES_PerformSearch1091<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1092: CUDA_DES_PerformSearch1092<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1093: CUDA_DES_PerformSearch1093<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1094: CUDA_DES_PerformSearch1094<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1095: CUDA_DES_PerformSearch1095<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1096: CUDA_DES_PerformSearch1096<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1097: CUDA_DES_PerformSearch1097<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1098: CUDA_DES_PerformSearch1098<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1099: CUDA_DES_PerformSearch1099<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1100: CUDA_DES_PerformSearch1100<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1101: CUDA_DES_PerformSearch1101<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1102: CUDA_DES_PerformSearch1102<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1103: CUDA_DES_PerformSearch1103<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1104: CUDA_DES_PerformSearch1104<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1105: CUDA_DES_PerformSearch1105<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1106: CUDA_DES_PerformSearch1106<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1107: CUDA_DES_PerformSearch1107<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1108: CUDA_DES_PerformSearch1108<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1109: CUDA_DES_PerformSearch1109<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1110: CUDA_DES_PerformSearch1110<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1111: CUDA_DES_PerformSearch1111<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1112: CUDA_DES_PerformSearch1112<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1113: CUDA_DES_PerformSearch1113<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1114: CUDA_DES_PerformSearch1114<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1115: CUDA_DES_PerformSearch1115<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1116: CUDA_DES_PerformSearch1116<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1117: CUDA_DES_PerformSearch1117<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1118: CUDA_DES_PerformSearch1118<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1119: CUDA_DES_PerformSearch1119<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1120: CUDA_DES_PerformSearch1120<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1121: CUDA_DES_PerformSearch1121<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1122: CUDA_DES_PerformSearch1122<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1123: CUDA_DES_PerformSearch1123<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1124: CUDA_DES_PerformSearch1124<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1125: CUDA_DES_PerformSearch1125<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1126: CUDA_DES_PerformSearch1126<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1127: CUDA_DES_PerformSearch1127<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1128: CUDA_DES_PerformSearch1128<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1129: CUDA_DES_PerformSearch1129<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1130: CUDA_DES_PerformSearch1130<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1131: CUDA_DES_PerformSearch1131<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1132: CUDA_DES_PerformSearch1132<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1133: CUDA_DES_PerformSearch1133<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1134: CUDA_DES_PerformSearch1134<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1135: CUDA_DES_PerformSearch1135<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1136: CUDA_DES_PerformSearch1136<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1137: CUDA_DES_PerformSearch1137<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1138: CUDA_DES_PerformSearch1138<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1139: CUDA_DES_PerformSearch1139<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1140: CUDA_DES_PerformSearch1140<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1141: CUDA_DES_PerformSearch1141<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1142: CUDA_DES_PerformSearch1142<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1143: CUDA_DES_PerformSearch1143<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1144: CUDA_DES_PerformSearch1144<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1145: CUDA_DES_PerformSearch1145<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1146: CUDA_DES_PerformSearch1146<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1147: CUDA_DES_PerformSearch1147<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1148: CUDA_DES_PerformSearch1148<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1149: CUDA_DES_PerformSearch1149<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1150: CUDA_DES_PerformSearch1150<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1151: CUDA_DES_PerformSearch1151<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1152: CUDA_DES_PerformSearch1152<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1153: CUDA_DES_PerformSearch1153<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1154: CUDA_DES_PerformSearch1154<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1155: CUDA_DES_PerformSearch1155<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1156: CUDA_DES_PerformSearch1156<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1157: CUDA_DES_PerformSearch1157<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1158: CUDA_DES_PerformSearch1158<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1159: CUDA_DES_PerformSearch1159<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1160: CUDA_DES_PerformSearch1160<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1161: CUDA_DES_PerformSearch1161<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1162: CUDA_DES_PerformSearch1162<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1163: CUDA_DES_PerformSearch1163<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1164: CUDA_DES_PerformSearch1164<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1165: CUDA_DES_PerformSearch1165<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1166: CUDA_DES_PerformSearch1166<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1167: CUDA_DES_PerformSearch1167<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1168: CUDA_DES_PerformSearch1168<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1169: CUDA_DES_PerformSearch1169<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1170: CUDA_DES_PerformSearch1170<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1171: CUDA_DES_PerformSearch1171<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1172: CUDA_DES_PerformSearch1172<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1173: CUDA_DES_PerformSearch1173<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1174: CUDA_DES_PerformSearch1174<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1175: CUDA_DES_PerformSearch1175<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1176: CUDA_DES_PerformSearch1176<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1177: CUDA_DES_PerformSearch1177<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1178: CUDA_DES_PerformSearch1178<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1179: CUDA_DES_PerformSearch1179<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1180: CUDA_DES_PerformSearch1180<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1181: CUDA_DES_PerformSearch1181<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1182: CUDA_DES_PerformSearch1182<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1183: CUDA_DES_PerformSearch1183<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1184: CUDA_DES_PerformSearch1184<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1185: CUDA_DES_PerformSearch1185<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1186: CUDA_DES_PerformSearch1186<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1187: CUDA_DES_PerformSearch1187<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1188: CUDA_DES_PerformSearch1188<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1189: CUDA_DES_PerformSearch1189<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1190: CUDA_DES_PerformSearch1190<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1191: CUDA_DES_PerformSearch1191<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1192: CUDA_DES_PerformSearch1192<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1193: CUDA_DES_PerformSearch1193<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1194: CUDA_DES_PerformSearch1194<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1195: CUDA_DES_PerformSearch1195<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1196: CUDA_DES_PerformSearch1196<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1197: CUDA_DES_PerformSearch1197<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1198: CUDA_DES_PerformSearch1198<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1199: CUDA_DES_PerformSearch1199<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1200: CUDA_DES_PerformSearch1200<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1201: CUDA_DES_PerformSearch1201<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1202: CUDA_DES_PerformSearch1202<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1203: CUDA_DES_PerformSearch1203<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1204: CUDA_DES_PerformSearch1204<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1205: CUDA_DES_PerformSearch1205<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1206: CUDA_DES_PerformSearch1206<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1207: CUDA_DES_PerformSearch1207<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1208: CUDA_DES_PerformSearch1208<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1209: CUDA_DES_PerformSearch1209<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1210: CUDA_DES_PerformSearch1210<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1211: CUDA_DES_PerformSearch1211<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1212: CUDA_DES_PerformSearch1212<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1213: CUDA_DES_PerformSearch1213<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1214: CUDA_DES_PerformSearch1214<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1215: CUDA_DES_PerformSearch1215<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1216: CUDA_DES_PerformSearch1216<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1217: CUDA_DES_PerformSearch1217<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1218: CUDA_DES_PerformSearch1218<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1219: CUDA_DES_PerformSearch1219<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1220: CUDA_DES_PerformSearch1220<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1221: CUDA_DES_PerformSearch1221<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1222: CUDA_DES_PerformSearch1222<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1223: CUDA_DES_PerformSearch1223<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1224: CUDA_DES_PerformSearch1224<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1225: CUDA_DES_PerformSearch1225<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1226: CUDA_DES_PerformSearch1226<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1227: CUDA_DES_PerformSearch1227<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1228: CUDA_DES_PerformSearch1228<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1229: CUDA_DES_PerformSearch1229<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1230: CUDA_DES_PerformSearch1230<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1231: CUDA_DES_PerformSearch1231<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1232: CUDA_DES_PerformSearch1232<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1233: CUDA_DES_PerformSearch1233<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1234: CUDA_DES_PerformSearch1234<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1235: CUDA_DES_PerformSearch1235<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1236: CUDA_DES_PerformSearch1236<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1237: CUDA_DES_PerformSearch1237<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1238: CUDA_DES_PerformSearch1238<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1239: CUDA_DES_PerformSearch1239<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1240: CUDA_DES_PerformSearch1240<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1241: CUDA_DES_PerformSearch1241<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1242: CUDA_DES_PerformSearch1242<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1243: CUDA_DES_PerformSearch1243<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1244: CUDA_DES_PerformSearch1244<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1245: CUDA_DES_PerformSearch1245<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1246: CUDA_DES_PerformSearch1246<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1247: CUDA_DES_PerformSearch1247<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1248: CUDA_DES_PerformSearch1248<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1249: CUDA_DES_PerformSearch1249<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1250: CUDA_DES_PerformSearch1250<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1251: CUDA_DES_PerformSearch1251<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1252: CUDA_DES_PerformSearch1252<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1253: CUDA_DES_PerformSearch1253<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1254: CUDA_DES_PerformSearch1254<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1255: CUDA_DES_PerformSearch1255<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1256: CUDA_DES_PerformSearch1256<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1257: CUDA_DES_PerformSearch1257<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1258: CUDA_DES_PerformSearch1258<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1259: CUDA_DES_PerformSearch1259<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1260: CUDA_DES_PerformSearch1260<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1261: CUDA_DES_PerformSearch1261<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1262: CUDA_DES_PerformSearch1262<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1263: CUDA_DES_PerformSearch1263<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1264: CUDA_DES_PerformSearch1264<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1265: CUDA_DES_PerformSearch1265<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1266: CUDA_DES_PerformSearch1266<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1267: CUDA_DES_PerformSearch1267<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1268: CUDA_DES_PerformSearch1268<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1269: CUDA_DES_PerformSearch1269<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1270: CUDA_DES_PerformSearch1270<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1271: CUDA_DES_PerformSearch1271<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1272: CUDA_DES_PerformSearch1272<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1273: CUDA_DES_PerformSearch1273<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1274: CUDA_DES_PerformSearch1274<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1275: CUDA_DES_PerformSearch1275<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1276: CUDA_DES_PerformSearch1276<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1277: CUDA_DES_PerformSearch1277<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1278: CUDA_DES_PerformSearch1278<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1279: CUDA_DES_PerformSearch1279<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1280: CUDA_DES_PerformSearch1280<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1281: CUDA_DES_PerformSearch1281<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1282: CUDA_DES_PerformSearch1282<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1283: CUDA_DES_PerformSearch1283<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1284: CUDA_DES_PerformSearch1284<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1285: CUDA_DES_PerformSearch1285<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1286: CUDA_DES_PerformSearch1286<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1287: CUDA_DES_PerformSearch1287<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1288: CUDA_DES_PerformSearch1288<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1289: CUDA_DES_PerformSearch1289<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1290: CUDA_DES_PerformSearch1290<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1291: CUDA_DES_PerformSearch1291<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1292: CUDA_DES_PerformSearch1292<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1293: CUDA_DES_PerformSearch1293<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1294: CUDA_DES_PerformSearch1294<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1295: CUDA_DES_PerformSearch1295<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1296: CUDA_DES_PerformSearch1296<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1297: CUDA_DES_PerformSearch1297<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1298: CUDA_DES_PerformSearch1298<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1299: CUDA_DES_PerformSearch1299<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1300: CUDA_DES_PerformSearch1300<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1301: CUDA_DES_PerformSearch1301<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1302: CUDA_DES_PerformSearch1302<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1303: CUDA_DES_PerformSearch1303<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1304: CUDA_DES_PerformSearch1304<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1305: CUDA_DES_PerformSearch1305<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1306: CUDA_DES_PerformSearch1306<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1307: CUDA_DES_PerformSearch1307<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1308: CUDA_DES_PerformSearch1308<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1309: CUDA_DES_PerformSearch1309<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1310: CUDA_DES_PerformSearch1310<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1311: CUDA_DES_PerformSearch1311<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1312: CUDA_DES_PerformSearch1312<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1313: CUDA_DES_PerformSearch1313<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1314: CUDA_DES_PerformSearch1314<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1315: CUDA_DES_PerformSearch1315<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1316: CUDA_DES_PerformSearch1316<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1317: CUDA_DES_PerformSearch1317<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1318: CUDA_DES_PerformSearch1318<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1319: CUDA_DES_PerformSearch1319<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1320: CUDA_DES_PerformSearch1320<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1321: CUDA_DES_PerformSearch1321<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1322: CUDA_DES_PerformSearch1322<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1323: CUDA_DES_PerformSearch1323<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1324: CUDA_DES_PerformSearch1324<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1325: CUDA_DES_PerformSearch1325<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1326: CUDA_DES_PerformSearch1326<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1327: CUDA_DES_PerformSearch1327<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1328: CUDA_DES_PerformSearch1328<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1329: CUDA_DES_PerformSearch1329<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1330: CUDA_DES_PerformSearch1330<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1331: CUDA_DES_PerformSearch1331<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1332: CUDA_DES_PerformSearch1332<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1333: CUDA_DES_PerformSearch1333<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1334: CUDA_DES_PerformSearch1334<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1335: CUDA_DES_PerformSearch1335<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1336: CUDA_DES_PerformSearch1336<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1337: CUDA_DES_PerformSearch1337<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1338: CUDA_DES_PerformSearch1338<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1339: CUDA_DES_PerformSearch1339<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1340: CUDA_DES_PerformSearch1340<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1341: CUDA_DES_PerformSearch1341<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1342: CUDA_DES_PerformSearch1342<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1343: CUDA_DES_PerformSearch1343<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1344: CUDA_DES_PerformSearch1344<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1345: CUDA_DES_PerformSearch1345<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1346: CUDA_DES_PerformSearch1346<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1347: CUDA_DES_PerformSearch1347<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1348: CUDA_DES_PerformSearch1348<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1349: CUDA_DES_PerformSearch1349<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1350: CUDA_DES_PerformSearch1350<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1351: CUDA_DES_PerformSearch1351<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1352: CUDA_DES_PerformSearch1352<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1353: CUDA_DES_PerformSearch1353<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1354: CUDA_DES_PerformSearch1354<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1355: CUDA_DES_PerformSearch1355<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1356: CUDA_DES_PerformSearch1356<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1357: CUDA_DES_PerformSearch1357<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1358: CUDA_DES_PerformSearch1358<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1359: CUDA_DES_PerformSearch1359<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1360: CUDA_DES_PerformSearch1360<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1361: CUDA_DES_PerformSearch1361<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1362: CUDA_DES_PerformSearch1362<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1363: CUDA_DES_PerformSearch1363<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1364: CUDA_DES_PerformSearch1364<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1365: CUDA_DES_PerformSearch1365<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1366: CUDA_DES_PerformSearch1366<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1367: CUDA_DES_PerformSearch1367<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1368: CUDA_DES_PerformSearch1368<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1369: CUDA_DES_PerformSearch1369<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1370: CUDA_DES_PerformSearch1370<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1371: CUDA_DES_PerformSearch1371<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1372: CUDA_DES_PerformSearch1372<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1373: CUDA_DES_PerformSearch1373<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1374: CUDA_DES_PerformSearch1374<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1375: CUDA_DES_PerformSearch1375<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1376: CUDA_DES_PerformSearch1376<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1377: CUDA_DES_PerformSearch1377<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1378: CUDA_DES_PerformSearch1378<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1379: CUDA_DES_PerformSearch1379<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1380: CUDA_DES_PerformSearch1380<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1381: CUDA_DES_PerformSearch1381<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1382: CUDA_DES_PerformSearch1382<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1383: CUDA_DES_PerformSearch1383<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1384: CUDA_DES_PerformSearch1384<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1385: CUDA_DES_PerformSearch1385<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1386: CUDA_DES_PerformSearch1386<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1387: CUDA_DES_PerformSearch1387<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1388: CUDA_DES_PerformSearch1388<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1389: CUDA_DES_PerformSearch1389<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1390: CUDA_DES_PerformSearch1390<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1391: CUDA_DES_PerformSearch1391<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1392: CUDA_DES_PerformSearch1392<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1393: CUDA_DES_PerformSearch1393<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1394: CUDA_DES_PerformSearch1394<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1395: CUDA_DES_PerformSearch1395<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1396: CUDA_DES_PerformSearch1396<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1397: CUDA_DES_PerformSearch1397<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1398: CUDA_DES_PerformSearch1398<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1399: CUDA_DES_PerformSearch1399<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1400: CUDA_DES_PerformSearch1400<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1401: CUDA_DES_PerformSearch1401<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1402: CUDA_DES_PerformSearch1402<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1403: CUDA_DES_PerformSearch1403<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1404: CUDA_DES_PerformSearch1404<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1405: CUDA_DES_PerformSearch1405<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1406: CUDA_DES_PerformSearch1406<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1407: CUDA_DES_PerformSearch1407<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1408: CUDA_DES_PerformSearch1408<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1409: CUDA_DES_PerformSearch1409<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1410: CUDA_DES_PerformSearch1410<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1411: CUDA_DES_PerformSearch1411<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1412: CUDA_DES_PerformSearch1412<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1413: CUDA_DES_PerformSearch1413<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1414: CUDA_DES_PerformSearch1414<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1415: CUDA_DES_PerformSearch1415<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1416: CUDA_DES_PerformSearch1416<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1417: CUDA_DES_PerformSearch1417<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1418: CUDA_DES_PerformSearch1418<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1419: CUDA_DES_PerformSearch1419<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1420: CUDA_DES_PerformSearch1420<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1421: CUDA_DES_PerformSearch1421<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1422: CUDA_DES_PerformSearch1422<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1423: CUDA_DES_PerformSearch1423<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1424: CUDA_DES_PerformSearch1424<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1425: CUDA_DES_PerformSearch1425<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1426: CUDA_DES_PerformSearch1426<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1427: CUDA_DES_PerformSearch1427<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1428: CUDA_DES_PerformSearch1428<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1429: CUDA_DES_PerformSearch1429<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1430: CUDA_DES_PerformSearch1430<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1431: CUDA_DES_PerformSearch1431<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1432: CUDA_DES_PerformSearch1432<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1433: CUDA_DES_PerformSearch1433<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1434: CUDA_DES_PerformSearch1434<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1435: CUDA_DES_PerformSearch1435<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1436: CUDA_DES_PerformSearch1436<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1437: CUDA_DES_PerformSearch1437<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1438: CUDA_DES_PerformSearch1438<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1439: CUDA_DES_PerformSearch1439<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1440: CUDA_DES_PerformSearch1440<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1441: CUDA_DES_PerformSearch1441<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1442: CUDA_DES_PerformSearch1442<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1443: CUDA_DES_PerformSearch1443<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1444: CUDA_DES_PerformSearch1444<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1445: CUDA_DES_PerformSearch1445<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1446: CUDA_DES_PerformSearch1446<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1447: CUDA_DES_PerformSearch1447<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1448: CUDA_DES_PerformSearch1448<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1449: CUDA_DES_PerformSearch1449<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1450: CUDA_DES_PerformSearch1450<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1451: CUDA_DES_PerformSearch1451<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1452: CUDA_DES_PerformSearch1452<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1453: CUDA_DES_PerformSearch1453<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1454: CUDA_DES_PerformSearch1454<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1455: CUDA_DES_PerformSearch1455<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1456: CUDA_DES_PerformSearch1456<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1457: CUDA_DES_PerformSearch1457<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1458: CUDA_DES_PerformSearch1458<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1459: CUDA_DES_PerformSearch1459<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1460: CUDA_DES_PerformSearch1460<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1461: CUDA_DES_PerformSearch1461<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1462: CUDA_DES_PerformSearch1462<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1463: CUDA_DES_PerformSearch1463<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1464: CUDA_DES_PerformSearch1464<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1465: CUDA_DES_PerformSearch1465<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1466: CUDA_DES_PerformSearch1466<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1467: CUDA_DES_PerformSearch1467<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1468: CUDA_DES_PerformSearch1468<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1469: CUDA_DES_PerformSearch1469<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1470: CUDA_DES_PerformSearch1470<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1471: CUDA_DES_PerformSearch1471<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1472: CUDA_DES_PerformSearch1472<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1473: CUDA_DES_PerformSearch1473<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1474: CUDA_DES_PerformSearch1474<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1475: CUDA_DES_PerformSearch1475<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1476: CUDA_DES_PerformSearch1476<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1477: CUDA_DES_PerformSearch1477<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1478: CUDA_DES_PerformSearch1478<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1479: CUDA_DES_PerformSearch1479<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1480: CUDA_DES_PerformSearch1480<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1481: CUDA_DES_PerformSearch1481<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1482: CUDA_DES_PerformSearch1482<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1483: CUDA_DES_PerformSearch1483<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1484: CUDA_DES_PerformSearch1484<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1485: CUDA_DES_PerformSearch1485<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1486: CUDA_DES_PerformSearch1486<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1487: CUDA_DES_PerformSearch1487<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1488: CUDA_DES_PerformSearch1488<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1489: CUDA_DES_PerformSearch1489<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1490: CUDA_DES_PerformSearch1490<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1491: CUDA_DES_PerformSearch1491<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1492: CUDA_DES_PerformSearch1492<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1493: CUDA_DES_PerformSearch1493<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1494: CUDA_DES_PerformSearch1494<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1495: CUDA_DES_PerformSearch1495<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1496: CUDA_DES_PerformSearch1496<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1497: CUDA_DES_PerformSearch1497<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1498: CUDA_DES_PerformSearch1498<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1499: CUDA_DES_PerformSearch1499<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1500: CUDA_DES_PerformSearch1500<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1501: CUDA_DES_PerformSearch1501<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1502: CUDA_DES_PerformSearch1502<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1503: CUDA_DES_PerformSearch1503<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1504: CUDA_DES_PerformSearch1504<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1505: CUDA_DES_PerformSearch1505<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1506: CUDA_DES_PerformSearch1506<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1507: CUDA_DES_PerformSearch1507<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1508: CUDA_DES_PerformSearch1508<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1509: CUDA_DES_PerformSearch1509<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1510: CUDA_DES_PerformSearch1510<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1511: CUDA_DES_PerformSearch1511<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1512: CUDA_DES_PerformSearch1512<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1513: CUDA_DES_PerformSearch1513<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1514: CUDA_DES_PerformSearch1514<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1515: CUDA_DES_PerformSearch1515<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1516: CUDA_DES_PerformSearch1516<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1517: CUDA_DES_PerformSearch1517<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1518: CUDA_DES_PerformSearch1518<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1519: CUDA_DES_PerformSearch1519<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1520: CUDA_DES_PerformSearch1520<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1521: CUDA_DES_PerformSearch1521<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1522: CUDA_DES_PerformSearch1522<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1523: CUDA_DES_PerformSearch1523<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1524: CUDA_DES_PerformSearch1524<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1525: CUDA_DES_PerformSearch1525<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1526: CUDA_DES_PerformSearch1526<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1527: CUDA_DES_PerformSearch1527<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1528: CUDA_DES_PerformSearch1528<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1529: CUDA_DES_PerformSearch1529<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1530: CUDA_DES_PerformSearch1530<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1531: CUDA_DES_PerformSearch1531<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1532: CUDA_DES_PerformSearch1532<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1533: CUDA_DES_PerformSearch1533<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1534: CUDA_DES_PerformSearch1534<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1535: CUDA_DES_PerformSearch1535<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
