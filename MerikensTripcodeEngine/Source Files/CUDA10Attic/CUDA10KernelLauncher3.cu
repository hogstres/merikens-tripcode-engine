// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 1536
#include "CUDA10Kernel.h"
#define SALT 1537
#include "CUDA10Kernel.h"
#define SALT 1538
#include "CUDA10Kernel.h"
#define SALT 1539
#include "CUDA10Kernel.h"
#define SALT 1540
#include "CUDA10Kernel.h"
#define SALT 1541
#include "CUDA10Kernel.h"
#define SALT 1542
#include "CUDA10Kernel.h"
#define SALT 1543
#include "CUDA10Kernel.h"
#define SALT 1544
#include "CUDA10Kernel.h"
#define SALT 1545
#include "CUDA10Kernel.h"
#define SALT 1546
#include "CUDA10Kernel.h"
#define SALT 1547
#include "CUDA10Kernel.h"
#define SALT 1548
#include "CUDA10Kernel.h"
#define SALT 1549
#include "CUDA10Kernel.h"
#define SALT 1550
#include "CUDA10Kernel.h"
#define SALT 1551
#include "CUDA10Kernel.h"
#define SALT 1552
#include "CUDA10Kernel.h"
#define SALT 1553
#include "CUDA10Kernel.h"
#define SALT 1554
#include "CUDA10Kernel.h"
#define SALT 1555
#include "CUDA10Kernel.h"
#define SALT 1556
#include "CUDA10Kernel.h"
#define SALT 1557
#include "CUDA10Kernel.h"
#define SALT 1558
#include "CUDA10Kernel.h"
#define SALT 1559
#include "CUDA10Kernel.h"
#define SALT 1560
#include "CUDA10Kernel.h"
#define SALT 1561
#include "CUDA10Kernel.h"
#define SALT 1562
#include "CUDA10Kernel.h"
#define SALT 1563
#include "CUDA10Kernel.h"
#define SALT 1564
#include "CUDA10Kernel.h"
#define SALT 1565
#include "CUDA10Kernel.h"
#define SALT 1566
#include "CUDA10Kernel.h"
#define SALT 1567
#include "CUDA10Kernel.h"
#define SALT 1568
#include "CUDA10Kernel.h"
#define SALT 1569
#include "CUDA10Kernel.h"
#define SALT 1570
#include "CUDA10Kernel.h"
#define SALT 1571
#include "CUDA10Kernel.h"
#define SALT 1572
#include "CUDA10Kernel.h"
#define SALT 1573
#include "CUDA10Kernel.h"
#define SALT 1574
#include "CUDA10Kernel.h"
#define SALT 1575
#include "CUDA10Kernel.h"
#define SALT 1576
#include "CUDA10Kernel.h"
#define SALT 1577
#include "CUDA10Kernel.h"
#define SALT 1578
#include "CUDA10Kernel.h"
#define SALT 1579
#include "CUDA10Kernel.h"
#define SALT 1580
#include "CUDA10Kernel.h"
#define SALT 1581
#include "CUDA10Kernel.h"
#define SALT 1582
#include "CUDA10Kernel.h"
#define SALT 1583
#include "CUDA10Kernel.h"
#define SALT 1584
#include "CUDA10Kernel.h"
#define SALT 1585
#include "CUDA10Kernel.h"
#define SALT 1586
#include "CUDA10Kernel.h"
#define SALT 1587
#include "CUDA10Kernel.h"
#define SALT 1588
#include "CUDA10Kernel.h"
#define SALT 1589
#include "CUDA10Kernel.h"
#define SALT 1590
#include "CUDA10Kernel.h"
#define SALT 1591
#include "CUDA10Kernel.h"
#define SALT 1592
#include "CUDA10Kernel.h"
#define SALT 1593
#include "CUDA10Kernel.h"
#define SALT 1594
#include "CUDA10Kernel.h"
#define SALT 1595
#include "CUDA10Kernel.h"
#define SALT 1596
#include "CUDA10Kernel.h"
#define SALT 1597
#include "CUDA10Kernel.h"
#define SALT 1598
#include "CUDA10Kernel.h"
#define SALT 1599
#include "CUDA10Kernel.h"
#define SALT 1600
#include "CUDA10Kernel.h"
#define SALT 1601
#include "CUDA10Kernel.h"
#define SALT 1602
#include "CUDA10Kernel.h"
#define SALT 1603
#include "CUDA10Kernel.h"
#define SALT 1604
#include "CUDA10Kernel.h"
#define SALT 1605
#include "CUDA10Kernel.h"
#define SALT 1606
#include "CUDA10Kernel.h"
#define SALT 1607
#include "CUDA10Kernel.h"
#define SALT 1608
#include "CUDA10Kernel.h"
#define SALT 1609
#include "CUDA10Kernel.h"
#define SALT 1610
#include "CUDA10Kernel.h"
#define SALT 1611
#include "CUDA10Kernel.h"
#define SALT 1612
#include "CUDA10Kernel.h"
#define SALT 1613
#include "CUDA10Kernel.h"
#define SALT 1614
#include "CUDA10Kernel.h"
#define SALT 1615
#include "CUDA10Kernel.h"
#define SALT 1616
#include "CUDA10Kernel.h"
#define SALT 1617
#include "CUDA10Kernel.h"
#define SALT 1618
#include "CUDA10Kernel.h"
#define SALT 1619
#include "CUDA10Kernel.h"
#define SALT 1620
#include "CUDA10Kernel.h"
#define SALT 1621
#include "CUDA10Kernel.h"
#define SALT 1622
#include "CUDA10Kernel.h"
#define SALT 1623
#include "CUDA10Kernel.h"
#define SALT 1624
#include "CUDA10Kernel.h"
#define SALT 1625
#include "CUDA10Kernel.h"
#define SALT 1626
#include "CUDA10Kernel.h"
#define SALT 1627
#include "CUDA10Kernel.h"
#define SALT 1628
#include "CUDA10Kernel.h"
#define SALT 1629
#include "CUDA10Kernel.h"
#define SALT 1630
#include "CUDA10Kernel.h"
#define SALT 1631
#include "CUDA10Kernel.h"
#define SALT 1632
#include "CUDA10Kernel.h"
#define SALT 1633
#include "CUDA10Kernel.h"
#define SALT 1634
#include "CUDA10Kernel.h"
#define SALT 1635
#include "CUDA10Kernel.h"
#define SALT 1636
#include "CUDA10Kernel.h"
#define SALT 1637
#include "CUDA10Kernel.h"
#define SALT 1638
#include "CUDA10Kernel.h"
#define SALT 1639
#include "CUDA10Kernel.h"
#define SALT 1640
#include "CUDA10Kernel.h"
#define SALT 1641
#include "CUDA10Kernel.h"
#define SALT 1642
#include "CUDA10Kernel.h"
#define SALT 1643
#include "CUDA10Kernel.h"
#define SALT 1644
#include "CUDA10Kernel.h"
#define SALT 1645
#include "CUDA10Kernel.h"
#define SALT 1646
#include "CUDA10Kernel.h"
#define SALT 1647
#include "CUDA10Kernel.h"
#define SALT 1648
#include "CUDA10Kernel.h"
#define SALT 1649
#include "CUDA10Kernel.h"
#define SALT 1650
#include "CUDA10Kernel.h"
#define SALT 1651
#include "CUDA10Kernel.h"
#define SALT 1652
#include "CUDA10Kernel.h"
#define SALT 1653
#include "CUDA10Kernel.h"
#define SALT 1654
#include "CUDA10Kernel.h"
#define SALT 1655
#include "CUDA10Kernel.h"
#define SALT 1656
#include "CUDA10Kernel.h"
#define SALT 1657
#include "CUDA10Kernel.h"
#define SALT 1658
#include "CUDA10Kernel.h"
#define SALT 1659
#include "CUDA10Kernel.h"
#define SALT 1660
#include "CUDA10Kernel.h"
#define SALT 1661
#include "CUDA10Kernel.h"
#define SALT 1662
#include "CUDA10Kernel.h"
#define SALT 1663
#include "CUDA10Kernel.h"
#define SALT 1664
#include "CUDA10Kernel.h"
#define SALT 1665
#include "CUDA10Kernel.h"
#define SALT 1666
#include "CUDA10Kernel.h"
#define SALT 1667
#include "CUDA10Kernel.h"
#define SALT 1668
#include "CUDA10Kernel.h"
#define SALT 1669
#include "CUDA10Kernel.h"
#define SALT 1670
#include "CUDA10Kernel.h"
#define SALT 1671
#include "CUDA10Kernel.h"
#define SALT 1672
#include "CUDA10Kernel.h"
#define SALT 1673
#include "CUDA10Kernel.h"
#define SALT 1674
#include "CUDA10Kernel.h"
#define SALT 1675
#include "CUDA10Kernel.h"
#define SALT 1676
#include "CUDA10Kernel.h"
#define SALT 1677
#include "CUDA10Kernel.h"
#define SALT 1678
#include "CUDA10Kernel.h"
#define SALT 1679
#include "CUDA10Kernel.h"
#define SALT 1680
#include "CUDA10Kernel.h"
#define SALT 1681
#include "CUDA10Kernel.h"
#define SALT 1682
#include "CUDA10Kernel.h"
#define SALT 1683
#include "CUDA10Kernel.h"
#define SALT 1684
#include "CUDA10Kernel.h"
#define SALT 1685
#include "CUDA10Kernel.h"
#define SALT 1686
#include "CUDA10Kernel.h"
#define SALT 1687
#include "CUDA10Kernel.h"
#define SALT 1688
#include "CUDA10Kernel.h"
#define SALT 1689
#include "CUDA10Kernel.h"
#define SALT 1690
#include "CUDA10Kernel.h"
#define SALT 1691
#include "CUDA10Kernel.h"
#define SALT 1692
#include "CUDA10Kernel.h"
#define SALT 1693
#include "CUDA10Kernel.h"
#define SALT 1694
#include "CUDA10Kernel.h"
#define SALT 1695
#include "CUDA10Kernel.h"
#define SALT 1696
#include "CUDA10Kernel.h"
#define SALT 1697
#include "CUDA10Kernel.h"
#define SALT 1698
#include "CUDA10Kernel.h"
#define SALT 1699
#include "CUDA10Kernel.h"
#define SALT 1700
#include "CUDA10Kernel.h"
#define SALT 1701
#include "CUDA10Kernel.h"
#define SALT 1702
#include "CUDA10Kernel.h"
#define SALT 1703
#include "CUDA10Kernel.h"
#define SALT 1704
#include "CUDA10Kernel.h"
#define SALT 1705
#include "CUDA10Kernel.h"
#define SALT 1706
#include "CUDA10Kernel.h"
#define SALT 1707
#include "CUDA10Kernel.h"
#define SALT 1708
#include "CUDA10Kernel.h"
#define SALT 1709
#include "CUDA10Kernel.h"
#define SALT 1710
#include "CUDA10Kernel.h"
#define SALT 1711
#include "CUDA10Kernel.h"
#define SALT 1712
#include "CUDA10Kernel.h"
#define SALT 1713
#include "CUDA10Kernel.h"
#define SALT 1714
#include "CUDA10Kernel.h"
#define SALT 1715
#include "CUDA10Kernel.h"
#define SALT 1716
#include "CUDA10Kernel.h"
#define SALT 1717
#include "CUDA10Kernel.h"
#define SALT 1718
#include "CUDA10Kernel.h"
#define SALT 1719
#include "CUDA10Kernel.h"
#define SALT 1720
#include "CUDA10Kernel.h"
#define SALT 1721
#include "CUDA10Kernel.h"
#define SALT 1722
#include "CUDA10Kernel.h"
#define SALT 1723
#include "CUDA10Kernel.h"
#define SALT 1724
#include "CUDA10Kernel.h"
#define SALT 1725
#include "CUDA10Kernel.h"
#define SALT 1726
#include "CUDA10Kernel.h"
#define SALT 1727
#include "CUDA10Kernel.h"
#define SALT 1728
#include "CUDA10Kernel.h"
#define SALT 1729
#include "CUDA10Kernel.h"
#define SALT 1730
#include "CUDA10Kernel.h"
#define SALT 1731
#include "CUDA10Kernel.h"
#define SALT 1732
#include "CUDA10Kernel.h"
#define SALT 1733
#include "CUDA10Kernel.h"
#define SALT 1734
#include "CUDA10Kernel.h"
#define SALT 1735
#include "CUDA10Kernel.h"
#define SALT 1736
#include "CUDA10Kernel.h"
#define SALT 1737
#include "CUDA10Kernel.h"
#define SALT 1738
#include "CUDA10Kernel.h"
#define SALT 1739
#include "CUDA10Kernel.h"
#define SALT 1740
#include "CUDA10Kernel.h"
#define SALT 1741
#include "CUDA10Kernel.h"
#define SALT 1742
#include "CUDA10Kernel.h"
#define SALT 1743
#include "CUDA10Kernel.h"
#define SALT 1744
#include "CUDA10Kernel.h"
#define SALT 1745
#include "CUDA10Kernel.h"
#define SALT 1746
#include "CUDA10Kernel.h"
#define SALT 1747
#include "CUDA10Kernel.h"
#define SALT 1748
#include "CUDA10Kernel.h"
#define SALT 1749
#include "CUDA10Kernel.h"
#define SALT 1750
#include "CUDA10Kernel.h"
#define SALT 1751
#include "CUDA10Kernel.h"
#define SALT 1752
#include "CUDA10Kernel.h"
#define SALT 1753
#include "CUDA10Kernel.h"
#define SALT 1754
#include "CUDA10Kernel.h"
#define SALT 1755
#include "CUDA10Kernel.h"
#define SALT 1756
#include "CUDA10Kernel.h"
#define SALT 1757
#include "CUDA10Kernel.h"
#define SALT 1758
#include "CUDA10Kernel.h"
#define SALT 1759
#include "CUDA10Kernel.h"
#define SALT 1760
#include "CUDA10Kernel.h"
#define SALT 1761
#include "CUDA10Kernel.h"
#define SALT 1762
#include "CUDA10Kernel.h"
#define SALT 1763
#include "CUDA10Kernel.h"
#define SALT 1764
#include "CUDA10Kernel.h"
#define SALT 1765
#include "CUDA10Kernel.h"
#define SALT 1766
#include "CUDA10Kernel.h"
#define SALT 1767
#include "CUDA10Kernel.h"
#define SALT 1768
#include "CUDA10Kernel.h"
#define SALT 1769
#include "CUDA10Kernel.h"
#define SALT 1770
#include "CUDA10Kernel.h"
#define SALT 1771
#include "CUDA10Kernel.h"
#define SALT 1772
#include "CUDA10Kernel.h"
#define SALT 1773
#include "CUDA10Kernel.h"
#define SALT 1774
#include "CUDA10Kernel.h"
#define SALT 1775
#include "CUDA10Kernel.h"
#define SALT 1776
#include "CUDA10Kernel.h"
#define SALT 1777
#include "CUDA10Kernel.h"
#define SALT 1778
#include "CUDA10Kernel.h"
#define SALT 1779
#include "CUDA10Kernel.h"
#define SALT 1780
#include "CUDA10Kernel.h"
#define SALT 1781
#include "CUDA10Kernel.h"
#define SALT 1782
#include "CUDA10Kernel.h"
#define SALT 1783
#include "CUDA10Kernel.h"
#define SALT 1784
#include "CUDA10Kernel.h"
#define SALT 1785
#include "CUDA10Kernel.h"
#define SALT 1786
#include "CUDA10Kernel.h"
#define SALT 1787
#include "CUDA10Kernel.h"
#define SALT 1788
#include "CUDA10Kernel.h"
#define SALT 1789
#include "CUDA10Kernel.h"
#define SALT 1790
#include "CUDA10Kernel.h"
#define SALT 1791
#include "CUDA10Kernel.h"
#define SALT 1792
#include "CUDA10Kernel.h"
#define SALT 1793
#include "CUDA10Kernel.h"
#define SALT 1794
#include "CUDA10Kernel.h"
#define SALT 1795
#include "CUDA10Kernel.h"
#define SALT 1796
#include "CUDA10Kernel.h"
#define SALT 1797
#include "CUDA10Kernel.h"
#define SALT 1798
#include "CUDA10Kernel.h"
#define SALT 1799
#include "CUDA10Kernel.h"
#define SALT 1800
#include "CUDA10Kernel.h"
#define SALT 1801
#include "CUDA10Kernel.h"
#define SALT 1802
#include "CUDA10Kernel.h"
#define SALT 1803
#include "CUDA10Kernel.h"
#define SALT 1804
#include "CUDA10Kernel.h"
#define SALT 1805
#include "CUDA10Kernel.h"
#define SALT 1806
#include "CUDA10Kernel.h"
#define SALT 1807
#include "CUDA10Kernel.h"
#define SALT 1808
#include "CUDA10Kernel.h"
#define SALT 1809
#include "CUDA10Kernel.h"
#define SALT 1810
#include "CUDA10Kernel.h"
#define SALT 1811
#include "CUDA10Kernel.h"
#define SALT 1812
#include "CUDA10Kernel.h"
#define SALT 1813
#include "CUDA10Kernel.h"
#define SALT 1814
#include "CUDA10Kernel.h"
#define SALT 1815
#include "CUDA10Kernel.h"
#define SALT 1816
#include "CUDA10Kernel.h"
#define SALT 1817
#include "CUDA10Kernel.h"
#define SALT 1818
#include "CUDA10Kernel.h"
#define SALT 1819
#include "CUDA10Kernel.h"
#define SALT 1820
#include "CUDA10Kernel.h"
#define SALT 1821
#include "CUDA10Kernel.h"
#define SALT 1822
#include "CUDA10Kernel.h"
#define SALT 1823
#include "CUDA10Kernel.h"
#define SALT 1824
#include "CUDA10Kernel.h"
#define SALT 1825
#include "CUDA10Kernel.h"
#define SALT 1826
#include "CUDA10Kernel.h"
#define SALT 1827
#include "CUDA10Kernel.h"
#define SALT 1828
#include "CUDA10Kernel.h"
#define SALT 1829
#include "CUDA10Kernel.h"
#define SALT 1830
#include "CUDA10Kernel.h"
#define SALT 1831
#include "CUDA10Kernel.h"
#define SALT 1832
#include "CUDA10Kernel.h"
#define SALT 1833
#include "CUDA10Kernel.h"
#define SALT 1834
#include "CUDA10Kernel.h"
#define SALT 1835
#include "CUDA10Kernel.h"
#define SALT 1836
#include "CUDA10Kernel.h"
#define SALT 1837
#include "CUDA10Kernel.h"
#define SALT 1838
#include "CUDA10Kernel.h"
#define SALT 1839
#include "CUDA10Kernel.h"
#define SALT 1840
#include "CUDA10Kernel.h"
#define SALT 1841
#include "CUDA10Kernel.h"
#define SALT 1842
#include "CUDA10Kernel.h"
#define SALT 1843
#include "CUDA10Kernel.h"
#define SALT 1844
#include "CUDA10Kernel.h"
#define SALT 1845
#include "CUDA10Kernel.h"
#define SALT 1846
#include "CUDA10Kernel.h"
#define SALT 1847
#include "CUDA10Kernel.h"
#define SALT 1848
#include "CUDA10Kernel.h"
#define SALT 1849
#include "CUDA10Kernel.h"
#define SALT 1850
#include "CUDA10Kernel.h"
#define SALT 1851
#include "CUDA10Kernel.h"
#define SALT 1852
#include "CUDA10Kernel.h"
#define SALT 1853
#include "CUDA10Kernel.h"
#define SALT 1854
#include "CUDA10Kernel.h"
#define SALT 1855
#include "CUDA10Kernel.h"
#define SALT 1856
#include "CUDA10Kernel.h"
#define SALT 1857
#include "CUDA10Kernel.h"
#define SALT 1858
#include "CUDA10Kernel.h"
#define SALT 1859
#include "CUDA10Kernel.h"
#define SALT 1860
#include "CUDA10Kernel.h"
#define SALT 1861
#include "CUDA10Kernel.h"
#define SALT 1862
#include "CUDA10Kernel.h"
#define SALT 1863
#include "CUDA10Kernel.h"
#define SALT 1864
#include "CUDA10Kernel.h"
#define SALT 1865
#include "CUDA10Kernel.h"
#define SALT 1866
#include "CUDA10Kernel.h"
#define SALT 1867
#include "CUDA10Kernel.h"
#define SALT 1868
#include "CUDA10Kernel.h"
#define SALT 1869
#include "CUDA10Kernel.h"
#define SALT 1870
#include "CUDA10Kernel.h"
#define SALT 1871
#include "CUDA10Kernel.h"
#define SALT 1872
#include "CUDA10Kernel.h"
#define SALT 1873
#include "CUDA10Kernel.h"
#define SALT 1874
#include "CUDA10Kernel.h"
#define SALT 1875
#include "CUDA10Kernel.h"
#define SALT 1876
#include "CUDA10Kernel.h"
#define SALT 1877
#include "CUDA10Kernel.h"
#define SALT 1878
#include "CUDA10Kernel.h"
#define SALT 1879
#include "CUDA10Kernel.h"
#define SALT 1880
#include "CUDA10Kernel.h"
#define SALT 1881
#include "CUDA10Kernel.h"
#define SALT 1882
#include "CUDA10Kernel.h"
#define SALT 1883
#include "CUDA10Kernel.h"
#define SALT 1884
#include "CUDA10Kernel.h"
#define SALT 1885
#include "CUDA10Kernel.h"
#define SALT 1886
#include "CUDA10Kernel.h"
#define SALT 1887
#include "CUDA10Kernel.h"
#define SALT 1888
#include "CUDA10Kernel.h"
#define SALT 1889
#include "CUDA10Kernel.h"
#define SALT 1890
#include "CUDA10Kernel.h"
#define SALT 1891
#include "CUDA10Kernel.h"
#define SALT 1892
#include "CUDA10Kernel.h"
#define SALT 1893
#include "CUDA10Kernel.h"
#define SALT 1894
#include "CUDA10Kernel.h"
#define SALT 1895
#include "CUDA10Kernel.h"
#define SALT 1896
#include "CUDA10Kernel.h"
#define SALT 1897
#include "CUDA10Kernel.h"
#define SALT 1898
#include "CUDA10Kernel.h"
#define SALT 1899
#include "CUDA10Kernel.h"
#define SALT 1900
#include "CUDA10Kernel.h"
#define SALT 1901
#include "CUDA10Kernel.h"
#define SALT 1902
#include "CUDA10Kernel.h"
#define SALT 1903
#include "CUDA10Kernel.h"
#define SALT 1904
#include "CUDA10Kernel.h"
#define SALT 1905
#include "CUDA10Kernel.h"
#define SALT 1906
#include "CUDA10Kernel.h"
#define SALT 1907
#include "CUDA10Kernel.h"
#define SALT 1908
#include "CUDA10Kernel.h"
#define SALT 1909
#include "CUDA10Kernel.h"
#define SALT 1910
#include "CUDA10Kernel.h"
#define SALT 1911
#include "CUDA10Kernel.h"
#define SALT 1912
#include "CUDA10Kernel.h"
#define SALT 1913
#include "CUDA10Kernel.h"
#define SALT 1914
#include "CUDA10Kernel.h"
#define SALT 1915
#include "CUDA10Kernel.h"
#define SALT 1916
#include "CUDA10Kernel.h"
#define SALT 1917
#include "CUDA10Kernel.h"
#define SALT 1918
#include "CUDA10Kernel.h"
#define SALT 1919
#include "CUDA10Kernel.h"
#define SALT 1920
#include "CUDA10Kernel.h"
#define SALT 1921
#include "CUDA10Kernel.h"
#define SALT 1922
#include "CUDA10Kernel.h"
#define SALT 1923
#include "CUDA10Kernel.h"
#define SALT 1924
#include "CUDA10Kernel.h"
#define SALT 1925
#include "CUDA10Kernel.h"
#define SALT 1926
#include "CUDA10Kernel.h"
#define SALT 1927
#include "CUDA10Kernel.h"
#define SALT 1928
#include "CUDA10Kernel.h"
#define SALT 1929
#include "CUDA10Kernel.h"
#define SALT 1930
#include "CUDA10Kernel.h"
#define SALT 1931
#include "CUDA10Kernel.h"
#define SALT 1932
#include "CUDA10Kernel.h"
#define SALT 1933
#include "CUDA10Kernel.h"
#define SALT 1934
#include "CUDA10Kernel.h"
#define SALT 1935
#include "CUDA10Kernel.h"
#define SALT 1936
#include "CUDA10Kernel.h"
#define SALT 1937
#include "CUDA10Kernel.h"
#define SALT 1938
#include "CUDA10Kernel.h"
#define SALT 1939
#include "CUDA10Kernel.h"
#define SALT 1940
#include "CUDA10Kernel.h"
#define SALT 1941
#include "CUDA10Kernel.h"
#define SALT 1942
#include "CUDA10Kernel.h"
#define SALT 1943
#include "CUDA10Kernel.h"
#define SALT 1944
#include "CUDA10Kernel.h"
#define SALT 1945
#include "CUDA10Kernel.h"
#define SALT 1946
#include "CUDA10Kernel.h"
#define SALT 1947
#include "CUDA10Kernel.h"
#define SALT 1948
#include "CUDA10Kernel.h"
#define SALT 1949
#include "CUDA10Kernel.h"
#define SALT 1950
#include "CUDA10Kernel.h"
#define SALT 1951
#include "CUDA10Kernel.h"
#define SALT 1952
#include "CUDA10Kernel.h"
#define SALT 1953
#include "CUDA10Kernel.h"
#define SALT 1954
#include "CUDA10Kernel.h"
#define SALT 1955
#include "CUDA10Kernel.h"
#define SALT 1956
#include "CUDA10Kernel.h"
#define SALT 1957
#include "CUDA10Kernel.h"
#define SALT 1958
#include "CUDA10Kernel.h"
#define SALT 1959
#include "CUDA10Kernel.h"
#define SALT 1960
#include "CUDA10Kernel.h"
#define SALT 1961
#include "CUDA10Kernel.h"
#define SALT 1962
#include "CUDA10Kernel.h"
#define SALT 1963
#include "CUDA10Kernel.h"
#define SALT 1964
#include "CUDA10Kernel.h"
#define SALT 1965
#include "CUDA10Kernel.h"
#define SALT 1966
#include "CUDA10Kernel.h"
#define SALT 1967
#include "CUDA10Kernel.h"
#define SALT 1968
#include "CUDA10Kernel.h"
#define SALT 1969
#include "CUDA10Kernel.h"
#define SALT 1970
#include "CUDA10Kernel.h"
#define SALT 1971
#include "CUDA10Kernel.h"
#define SALT 1972
#include "CUDA10Kernel.h"
#define SALT 1973
#include "CUDA10Kernel.h"
#define SALT 1974
#include "CUDA10Kernel.h"
#define SALT 1975
#include "CUDA10Kernel.h"
#define SALT 1976
#include "CUDA10Kernel.h"
#define SALT 1977
#include "CUDA10Kernel.h"
#define SALT 1978
#include "CUDA10Kernel.h"
#define SALT 1979
#include "CUDA10Kernel.h"
#define SALT 1980
#include "CUDA10Kernel.h"
#define SALT 1981
#include "CUDA10Kernel.h"
#define SALT 1982
#include "CUDA10Kernel.h"
#define SALT 1983
#include "CUDA10Kernel.h"
#define SALT 1984
#include "CUDA10Kernel.h"
#define SALT 1985
#include "CUDA10Kernel.h"
#define SALT 1986
#include "CUDA10Kernel.h"
#define SALT 1987
#include "CUDA10Kernel.h"
#define SALT 1988
#include "CUDA10Kernel.h"
#define SALT 1989
#include "CUDA10Kernel.h"
#define SALT 1990
#include "CUDA10Kernel.h"
#define SALT 1991
#include "CUDA10Kernel.h"
#define SALT 1992
#include "CUDA10Kernel.h"
#define SALT 1993
#include "CUDA10Kernel.h"
#define SALT 1994
#include "CUDA10Kernel.h"
#define SALT 1995
#include "CUDA10Kernel.h"
#define SALT 1996
#include "CUDA10Kernel.h"
#define SALT 1997
#include "CUDA10Kernel.h"
#define SALT 1998
#include "CUDA10Kernel.h"
#define SALT 1999
#include "CUDA10Kernel.h"
#define SALT 2000
#include "CUDA10Kernel.h"
#define SALT 2001
#include "CUDA10Kernel.h"
#define SALT 2002
#include "CUDA10Kernel.h"
#define SALT 2003
#include "CUDA10Kernel.h"
#define SALT 2004
#include "CUDA10Kernel.h"
#define SALT 2005
#include "CUDA10Kernel.h"
#define SALT 2006
#include "CUDA10Kernel.h"
#define SALT 2007
#include "CUDA10Kernel.h"
#define SALT 2008
#include "CUDA10Kernel.h"
#define SALT 2009
#include "CUDA10Kernel.h"
#define SALT 2010
#include "CUDA10Kernel.h"
#define SALT 2011
#include "CUDA10Kernel.h"
#define SALT 2012
#include "CUDA10Kernel.h"
#define SALT 2013
#include "CUDA10Kernel.h"
#define SALT 2014
#include "CUDA10Kernel.h"
#define SALT 2015
#include "CUDA10Kernel.h"
#define SALT 2016
#include "CUDA10Kernel.h"
#define SALT 2017
#include "CUDA10Kernel.h"
#define SALT 2018
#include "CUDA10Kernel.h"
#define SALT 2019
#include "CUDA10Kernel.h"
#define SALT 2020
#include "CUDA10Kernel.h"
#define SALT 2021
#include "CUDA10Kernel.h"
#define SALT 2022
#include "CUDA10Kernel.h"
#define SALT 2023
#include "CUDA10Kernel.h"
#define SALT 2024
#include "CUDA10Kernel.h"
#define SALT 2025
#include "CUDA10Kernel.h"
#define SALT 2026
#include "CUDA10Kernel.h"
#define SALT 2027
#include "CUDA10Kernel.h"
#define SALT 2028
#include "CUDA10Kernel.h"
#define SALT 2029
#include "CUDA10Kernel.h"
#define SALT 2030
#include "CUDA10Kernel.h"
#define SALT 2031
#include "CUDA10Kernel.h"
#define SALT 2032
#include "CUDA10Kernel.h"
#define SALT 2033
#include "CUDA10Kernel.h"
#define SALT 2034
#include "CUDA10Kernel.h"
#define SALT 2035
#include "CUDA10Kernel.h"
#define SALT 2036
#include "CUDA10Kernel.h"
#define SALT 2037
#include "CUDA10Kernel.h"
#define SALT 2038
#include "CUDA10Kernel.h"
#define SALT 2039
#include "CUDA10Kernel.h"
#define SALT 2040
#include "CUDA10Kernel.h"
#define SALT 2041
#include "CUDA10Kernel.h"
#define SALT 2042
#include "CUDA10Kernel.h"
#define SALT 2043
#include "CUDA10Kernel.h"
#define SALT 2044
#include "CUDA10Kernel.h"
#define SALT 2045
#include "CUDA10Kernel.h"
#define SALT 2046
#include "CUDA10Kernel.h"
#define SALT 2047
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher3()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel3(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 1536: CUDA_DES_PerformSearch1536<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1537: CUDA_DES_PerformSearch1537<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1538: CUDA_DES_PerformSearch1538<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1539: CUDA_DES_PerformSearch1539<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1540: CUDA_DES_PerformSearch1540<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1541: CUDA_DES_PerformSearch1541<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1542: CUDA_DES_PerformSearch1542<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1543: CUDA_DES_PerformSearch1543<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1544: CUDA_DES_PerformSearch1544<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1545: CUDA_DES_PerformSearch1545<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1546: CUDA_DES_PerformSearch1546<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1547: CUDA_DES_PerformSearch1547<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1548: CUDA_DES_PerformSearch1548<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1549: CUDA_DES_PerformSearch1549<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1550: CUDA_DES_PerformSearch1550<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1551: CUDA_DES_PerformSearch1551<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1552: CUDA_DES_PerformSearch1552<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1553: CUDA_DES_PerformSearch1553<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1554: CUDA_DES_PerformSearch1554<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1555: CUDA_DES_PerformSearch1555<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1556: CUDA_DES_PerformSearch1556<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1557: CUDA_DES_PerformSearch1557<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1558: CUDA_DES_PerformSearch1558<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1559: CUDA_DES_PerformSearch1559<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1560: CUDA_DES_PerformSearch1560<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1561: CUDA_DES_PerformSearch1561<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1562: CUDA_DES_PerformSearch1562<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1563: CUDA_DES_PerformSearch1563<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1564: CUDA_DES_PerformSearch1564<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1565: CUDA_DES_PerformSearch1565<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1566: CUDA_DES_PerformSearch1566<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1567: CUDA_DES_PerformSearch1567<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1568: CUDA_DES_PerformSearch1568<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1569: CUDA_DES_PerformSearch1569<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1570: CUDA_DES_PerformSearch1570<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1571: CUDA_DES_PerformSearch1571<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1572: CUDA_DES_PerformSearch1572<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1573: CUDA_DES_PerformSearch1573<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1574: CUDA_DES_PerformSearch1574<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1575: CUDA_DES_PerformSearch1575<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1576: CUDA_DES_PerformSearch1576<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1577: CUDA_DES_PerformSearch1577<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1578: CUDA_DES_PerformSearch1578<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1579: CUDA_DES_PerformSearch1579<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1580: CUDA_DES_PerformSearch1580<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1581: CUDA_DES_PerformSearch1581<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1582: CUDA_DES_PerformSearch1582<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1583: CUDA_DES_PerformSearch1583<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1584: CUDA_DES_PerformSearch1584<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1585: CUDA_DES_PerformSearch1585<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1586: CUDA_DES_PerformSearch1586<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1587: CUDA_DES_PerformSearch1587<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1588: CUDA_DES_PerformSearch1588<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1589: CUDA_DES_PerformSearch1589<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1590: CUDA_DES_PerformSearch1590<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1591: CUDA_DES_PerformSearch1591<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1592: CUDA_DES_PerformSearch1592<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1593: CUDA_DES_PerformSearch1593<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1594: CUDA_DES_PerformSearch1594<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1595: CUDA_DES_PerformSearch1595<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1596: CUDA_DES_PerformSearch1596<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1597: CUDA_DES_PerformSearch1597<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1598: CUDA_DES_PerformSearch1598<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1599: CUDA_DES_PerformSearch1599<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1600: CUDA_DES_PerformSearch1600<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1601: CUDA_DES_PerformSearch1601<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1602: CUDA_DES_PerformSearch1602<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1603: CUDA_DES_PerformSearch1603<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1604: CUDA_DES_PerformSearch1604<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1605: CUDA_DES_PerformSearch1605<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1606: CUDA_DES_PerformSearch1606<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1607: CUDA_DES_PerformSearch1607<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1608: CUDA_DES_PerformSearch1608<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1609: CUDA_DES_PerformSearch1609<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1610: CUDA_DES_PerformSearch1610<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1611: CUDA_DES_PerformSearch1611<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1612: CUDA_DES_PerformSearch1612<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1613: CUDA_DES_PerformSearch1613<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1614: CUDA_DES_PerformSearch1614<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1615: CUDA_DES_PerformSearch1615<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1616: CUDA_DES_PerformSearch1616<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1617: CUDA_DES_PerformSearch1617<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1618: CUDA_DES_PerformSearch1618<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1619: CUDA_DES_PerformSearch1619<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1620: CUDA_DES_PerformSearch1620<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1621: CUDA_DES_PerformSearch1621<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1622: CUDA_DES_PerformSearch1622<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1623: CUDA_DES_PerformSearch1623<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1624: CUDA_DES_PerformSearch1624<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1625: CUDA_DES_PerformSearch1625<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1626: CUDA_DES_PerformSearch1626<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1627: CUDA_DES_PerformSearch1627<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1628: CUDA_DES_PerformSearch1628<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1629: CUDA_DES_PerformSearch1629<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1630: CUDA_DES_PerformSearch1630<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1631: CUDA_DES_PerformSearch1631<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1632: CUDA_DES_PerformSearch1632<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1633: CUDA_DES_PerformSearch1633<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1634: CUDA_DES_PerformSearch1634<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1635: CUDA_DES_PerformSearch1635<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1636: CUDA_DES_PerformSearch1636<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1637: CUDA_DES_PerformSearch1637<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1638: CUDA_DES_PerformSearch1638<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1639: CUDA_DES_PerformSearch1639<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1640: CUDA_DES_PerformSearch1640<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1641: CUDA_DES_PerformSearch1641<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1642: CUDA_DES_PerformSearch1642<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1643: CUDA_DES_PerformSearch1643<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1644: CUDA_DES_PerformSearch1644<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1645: CUDA_DES_PerformSearch1645<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1646: CUDA_DES_PerformSearch1646<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1647: CUDA_DES_PerformSearch1647<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1648: CUDA_DES_PerformSearch1648<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1649: CUDA_DES_PerformSearch1649<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1650: CUDA_DES_PerformSearch1650<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1651: CUDA_DES_PerformSearch1651<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1652: CUDA_DES_PerformSearch1652<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1653: CUDA_DES_PerformSearch1653<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1654: CUDA_DES_PerformSearch1654<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1655: CUDA_DES_PerformSearch1655<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1656: CUDA_DES_PerformSearch1656<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1657: CUDA_DES_PerformSearch1657<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1658: CUDA_DES_PerformSearch1658<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1659: CUDA_DES_PerformSearch1659<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1660: CUDA_DES_PerformSearch1660<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1661: CUDA_DES_PerformSearch1661<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1662: CUDA_DES_PerformSearch1662<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1663: CUDA_DES_PerformSearch1663<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1664: CUDA_DES_PerformSearch1664<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1665: CUDA_DES_PerformSearch1665<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1666: CUDA_DES_PerformSearch1666<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1667: CUDA_DES_PerformSearch1667<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1668: CUDA_DES_PerformSearch1668<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1669: CUDA_DES_PerformSearch1669<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1670: CUDA_DES_PerformSearch1670<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1671: CUDA_DES_PerformSearch1671<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1672: CUDA_DES_PerformSearch1672<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1673: CUDA_DES_PerformSearch1673<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1674: CUDA_DES_PerformSearch1674<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1675: CUDA_DES_PerformSearch1675<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1676: CUDA_DES_PerformSearch1676<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1677: CUDA_DES_PerformSearch1677<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1678: CUDA_DES_PerformSearch1678<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1679: CUDA_DES_PerformSearch1679<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1680: CUDA_DES_PerformSearch1680<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1681: CUDA_DES_PerformSearch1681<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1682: CUDA_DES_PerformSearch1682<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1683: CUDA_DES_PerformSearch1683<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1684: CUDA_DES_PerformSearch1684<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1685: CUDA_DES_PerformSearch1685<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1686: CUDA_DES_PerformSearch1686<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1687: CUDA_DES_PerformSearch1687<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1688: CUDA_DES_PerformSearch1688<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1689: CUDA_DES_PerformSearch1689<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1690: CUDA_DES_PerformSearch1690<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1691: CUDA_DES_PerformSearch1691<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1692: CUDA_DES_PerformSearch1692<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1693: CUDA_DES_PerformSearch1693<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1694: CUDA_DES_PerformSearch1694<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1695: CUDA_DES_PerformSearch1695<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1696: CUDA_DES_PerformSearch1696<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1697: CUDA_DES_PerformSearch1697<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1698: CUDA_DES_PerformSearch1698<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1699: CUDA_DES_PerformSearch1699<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1700: CUDA_DES_PerformSearch1700<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1701: CUDA_DES_PerformSearch1701<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1702: CUDA_DES_PerformSearch1702<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1703: CUDA_DES_PerformSearch1703<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1704: CUDA_DES_PerformSearch1704<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1705: CUDA_DES_PerformSearch1705<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1706: CUDA_DES_PerformSearch1706<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1707: CUDA_DES_PerformSearch1707<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1708: CUDA_DES_PerformSearch1708<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1709: CUDA_DES_PerformSearch1709<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1710: CUDA_DES_PerformSearch1710<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1711: CUDA_DES_PerformSearch1711<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1712: CUDA_DES_PerformSearch1712<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1713: CUDA_DES_PerformSearch1713<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1714: CUDA_DES_PerformSearch1714<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1715: CUDA_DES_PerformSearch1715<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1716: CUDA_DES_PerformSearch1716<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1717: CUDA_DES_PerformSearch1717<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1718: CUDA_DES_PerformSearch1718<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1719: CUDA_DES_PerformSearch1719<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1720: CUDA_DES_PerformSearch1720<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1721: CUDA_DES_PerformSearch1721<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1722: CUDA_DES_PerformSearch1722<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1723: CUDA_DES_PerformSearch1723<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1724: CUDA_DES_PerformSearch1724<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1725: CUDA_DES_PerformSearch1725<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1726: CUDA_DES_PerformSearch1726<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1727: CUDA_DES_PerformSearch1727<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1728: CUDA_DES_PerformSearch1728<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1729: CUDA_DES_PerformSearch1729<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1730: CUDA_DES_PerformSearch1730<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1731: CUDA_DES_PerformSearch1731<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1732: CUDA_DES_PerformSearch1732<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1733: CUDA_DES_PerformSearch1733<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1734: CUDA_DES_PerformSearch1734<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1735: CUDA_DES_PerformSearch1735<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1736: CUDA_DES_PerformSearch1736<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1737: CUDA_DES_PerformSearch1737<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1738: CUDA_DES_PerformSearch1738<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1739: CUDA_DES_PerformSearch1739<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1740: CUDA_DES_PerformSearch1740<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1741: CUDA_DES_PerformSearch1741<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1742: CUDA_DES_PerformSearch1742<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1743: CUDA_DES_PerformSearch1743<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1744: CUDA_DES_PerformSearch1744<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1745: CUDA_DES_PerformSearch1745<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1746: CUDA_DES_PerformSearch1746<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1747: CUDA_DES_PerformSearch1747<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1748: CUDA_DES_PerformSearch1748<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1749: CUDA_DES_PerformSearch1749<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1750: CUDA_DES_PerformSearch1750<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1751: CUDA_DES_PerformSearch1751<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1752: CUDA_DES_PerformSearch1752<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1753: CUDA_DES_PerformSearch1753<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1754: CUDA_DES_PerformSearch1754<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1755: CUDA_DES_PerformSearch1755<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1756: CUDA_DES_PerformSearch1756<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1757: CUDA_DES_PerformSearch1757<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1758: CUDA_DES_PerformSearch1758<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1759: CUDA_DES_PerformSearch1759<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1760: CUDA_DES_PerformSearch1760<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1761: CUDA_DES_PerformSearch1761<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1762: CUDA_DES_PerformSearch1762<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1763: CUDA_DES_PerformSearch1763<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1764: CUDA_DES_PerformSearch1764<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1765: CUDA_DES_PerformSearch1765<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1766: CUDA_DES_PerformSearch1766<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1767: CUDA_DES_PerformSearch1767<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1768: CUDA_DES_PerformSearch1768<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1769: CUDA_DES_PerformSearch1769<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1770: CUDA_DES_PerformSearch1770<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1771: CUDA_DES_PerformSearch1771<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1772: CUDA_DES_PerformSearch1772<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1773: CUDA_DES_PerformSearch1773<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1774: CUDA_DES_PerformSearch1774<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1775: CUDA_DES_PerformSearch1775<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1776: CUDA_DES_PerformSearch1776<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1777: CUDA_DES_PerformSearch1777<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1778: CUDA_DES_PerformSearch1778<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1779: CUDA_DES_PerformSearch1779<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1780: CUDA_DES_PerformSearch1780<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1781: CUDA_DES_PerformSearch1781<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1782: CUDA_DES_PerformSearch1782<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1783: CUDA_DES_PerformSearch1783<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1784: CUDA_DES_PerformSearch1784<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1785: CUDA_DES_PerformSearch1785<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1786: CUDA_DES_PerformSearch1786<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1787: CUDA_DES_PerformSearch1787<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1788: CUDA_DES_PerformSearch1788<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1789: CUDA_DES_PerformSearch1789<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1790: CUDA_DES_PerformSearch1790<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1791: CUDA_DES_PerformSearch1791<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1792: CUDA_DES_PerformSearch1792<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1793: CUDA_DES_PerformSearch1793<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1794: CUDA_DES_PerformSearch1794<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1795: CUDA_DES_PerformSearch1795<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1796: CUDA_DES_PerformSearch1796<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1797: CUDA_DES_PerformSearch1797<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1798: CUDA_DES_PerformSearch1798<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1799: CUDA_DES_PerformSearch1799<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1800: CUDA_DES_PerformSearch1800<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1801: CUDA_DES_PerformSearch1801<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1802: CUDA_DES_PerformSearch1802<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1803: CUDA_DES_PerformSearch1803<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1804: CUDA_DES_PerformSearch1804<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1805: CUDA_DES_PerformSearch1805<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1806: CUDA_DES_PerformSearch1806<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1807: CUDA_DES_PerformSearch1807<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1808: CUDA_DES_PerformSearch1808<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1809: CUDA_DES_PerformSearch1809<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1810: CUDA_DES_PerformSearch1810<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1811: CUDA_DES_PerformSearch1811<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1812: CUDA_DES_PerformSearch1812<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1813: CUDA_DES_PerformSearch1813<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1814: CUDA_DES_PerformSearch1814<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1815: CUDA_DES_PerformSearch1815<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1816: CUDA_DES_PerformSearch1816<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1817: CUDA_DES_PerformSearch1817<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1818: CUDA_DES_PerformSearch1818<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1819: CUDA_DES_PerformSearch1819<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1820: CUDA_DES_PerformSearch1820<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1821: CUDA_DES_PerformSearch1821<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1822: CUDA_DES_PerformSearch1822<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1823: CUDA_DES_PerformSearch1823<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1824: CUDA_DES_PerformSearch1824<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1825: CUDA_DES_PerformSearch1825<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1826: CUDA_DES_PerformSearch1826<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1827: CUDA_DES_PerformSearch1827<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1828: CUDA_DES_PerformSearch1828<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1829: CUDA_DES_PerformSearch1829<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1830: CUDA_DES_PerformSearch1830<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1831: CUDA_DES_PerformSearch1831<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1832: CUDA_DES_PerformSearch1832<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1833: CUDA_DES_PerformSearch1833<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1834: CUDA_DES_PerformSearch1834<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1835: CUDA_DES_PerformSearch1835<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1836: CUDA_DES_PerformSearch1836<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1837: CUDA_DES_PerformSearch1837<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1838: CUDA_DES_PerformSearch1838<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1839: CUDA_DES_PerformSearch1839<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1840: CUDA_DES_PerformSearch1840<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1841: CUDA_DES_PerformSearch1841<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1842: CUDA_DES_PerformSearch1842<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1843: CUDA_DES_PerformSearch1843<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1844: CUDA_DES_PerformSearch1844<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1845: CUDA_DES_PerformSearch1845<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1846: CUDA_DES_PerformSearch1846<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1847: CUDA_DES_PerformSearch1847<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1848: CUDA_DES_PerformSearch1848<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1849: CUDA_DES_PerformSearch1849<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1850: CUDA_DES_PerformSearch1850<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1851: CUDA_DES_PerformSearch1851<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1852: CUDA_DES_PerformSearch1852<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1853: CUDA_DES_PerformSearch1853<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1854: CUDA_DES_PerformSearch1854<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1855: CUDA_DES_PerformSearch1855<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1856: CUDA_DES_PerformSearch1856<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1857: CUDA_DES_PerformSearch1857<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1858: CUDA_DES_PerformSearch1858<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1859: CUDA_DES_PerformSearch1859<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1860: CUDA_DES_PerformSearch1860<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1861: CUDA_DES_PerformSearch1861<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1862: CUDA_DES_PerformSearch1862<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1863: CUDA_DES_PerformSearch1863<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1864: CUDA_DES_PerformSearch1864<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1865: CUDA_DES_PerformSearch1865<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1866: CUDA_DES_PerformSearch1866<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1867: CUDA_DES_PerformSearch1867<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1868: CUDA_DES_PerformSearch1868<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1869: CUDA_DES_PerformSearch1869<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1870: CUDA_DES_PerformSearch1870<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1871: CUDA_DES_PerformSearch1871<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1872: CUDA_DES_PerformSearch1872<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1873: CUDA_DES_PerformSearch1873<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1874: CUDA_DES_PerformSearch1874<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1875: CUDA_DES_PerformSearch1875<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1876: CUDA_DES_PerformSearch1876<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1877: CUDA_DES_PerformSearch1877<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1878: CUDA_DES_PerformSearch1878<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1879: CUDA_DES_PerformSearch1879<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1880: CUDA_DES_PerformSearch1880<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1881: CUDA_DES_PerformSearch1881<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1882: CUDA_DES_PerformSearch1882<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1883: CUDA_DES_PerformSearch1883<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1884: CUDA_DES_PerformSearch1884<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1885: CUDA_DES_PerformSearch1885<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1886: CUDA_DES_PerformSearch1886<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1887: CUDA_DES_PerformSearch1887<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1888: CUDA_DES_PerformSearch1888<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1889: CUDA_DES_PerformSearch1889<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1890: CUDA_DES_PerformSearch1890<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1891: CUDA_DES_PerformSearch1891<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1892: CUDA_DES_PerformSearch1892<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1893: CUDA_DES_PerformSearch1893<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1894: CUDA_DES_PerformSearch1894<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1895: CUDA_DES_PerformSearch1895<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1896: CUDA_DES_PerformSearch1896<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1897: CUDA_DES_PerformSearch1897<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1898: CUDA_DES_PerformSearch1898<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1899: CUDA_DES_PerformSearch1899<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1900: CUDA_DES_PerformSearch1900<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1901: CUDA_DES_PerformSearch1901<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1902: CUDA_DES_PerformSearch1902<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1903: CUDA_DES_PerformSearch1903<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1904: CUDA_DES_PerformSearch1904<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1905: CUDA_DES_PerformSearch1905<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1906: CUDA_DES_PerformSearch1906<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1907: CUDA_DES_PerformSearch1907<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1908: CUDA_DES_PerformSearch1908<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1909: CUDA_DES_PerformSearch1909<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1910: CUDA_DES_PerformSearch1910<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1911: CUDA_DES_PerformSearch1911<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1912: CUDA_DES_PerformSearch1912<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1913: CUDA_DES_PerformSearch1913<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1914: CUDA_DES_PerformSearch1914<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1915: CUDA_DES_PerformSearch1915<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1916: CUDA_DES_PerformSearch1916<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1917: CUDA_DES_PerformSearch1917<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1918: CUDA_DES_PerformSearch1918<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1919: CUDA_DES_PerformSearch1919<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1920: CUDA_DES_PerformSearch1920<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1921: CUDA_DES_PerformSearch1921<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1922: CUDA_DES_PerformSearch1922<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1923: CUDA_DES_PerformSearch1923<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1924: CUDA_DES_PerformSearch1924<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1925: CUDA_DES_PerformSearch1925<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1926: CUDA_DES_PerformSearch1926<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1927: CUDA_DES_PerformSearch1927<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1928: CUDA_DES_PerformSearch1928<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1929: CUDA_DES_PerformSearch1929<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1930: CUDA_DES_PerformSearch1930<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1931: CUDA_DES_PerformSearch1931<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1932: CUDA_DES_PerformSearch1932<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1933: CUDA_DES_PerformSearch1933<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1934: CUDA_DES_PerformSearch1934<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1935: CUDA_DES_PerformSearch1935<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1936: CUDA_DES_PerformSearch1936<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1937: CUDA_DES_PerformSearch1937<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1938: CUDA_DES_PerformSearch1938<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1939: CUDA_DES_PerformSearch1939<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1940: CUDA_DES_PerformSearch1940<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1941: CUDA_DES_PerformSearch1941<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1942: CUDA_DES_PerformSearch1942<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1943: CUDA_DES_PerformSearch1943<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1944: CUDA_DES_PerformSearch1944<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1945: CUDA_DES_PerformSearch1945<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1946: CUDA_DES_PerformSearch1946<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1947: CUDA_DES_PerformSearch1947<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1948: CUDA_DES_PerformSearch1948<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1949: CUDA_DES_PerformSearch1949<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1950: CUDA_DES_PerformSearch1950<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1951: CUDA_DES_PerformSearch1951<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1952: CUDA_DES_PerformSearch1952<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1953: CUDA_DES_PerformSearch1953<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1954: CUDA_DES_PerformSearch1954<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1955: CUDA_DES_PerformSearch1955<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1956: CUDA_DES_PerformSearch1956<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1957: CUDA_DES_PerformSearch1957<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1958: CUDA_DES_PerformSearch1958<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1959: CUDA_DES_PerformSearch1959<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1960: CUDA_DES_PerformSearch1960<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1961: CUDA_DES_PerformSearch1961<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1962: CUDA_DES_PerformSearch1962<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1963: CUDA_DES_PerformSearch1963<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1964: CUDA_DES_PerformSearch1964<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1965: CUDA_DES_PerformSearch1965<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1966: CUDA_DES_PerformSearch1966<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1967: CUDA_DES_PerformSearch1967<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1968: CUDA_DES_PerformSearch1968<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1969: CUDA_DES_PerformSearch1969<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1970: CUDA_DES_PerformSearch1970<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1971: CUDA_DES_PerformSearch1971<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1972: CUDA_DES_PerformSearch1972<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1973: CUDA_DES_PerformSearch1973<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1974: CUDA_DES_PerformSearch1974<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1975: CUDA_DES_PerformSearch1975<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1976: CUDA_DES_PerformSearch1976<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1977: CUDA_DES_PerformSearch1977<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1978: CUDA_DES_PerformSearch1978<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1979: CUDA_DES_PerformSearch1979<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1980: CUDA_DES_PerformSearch1980<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1981: CUDA_DES_PerformSearch1981<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1982: CUDA_DES_PerformSearch1982<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1983: CUDA_DES_PerformSearch1983<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1984: CUDA_DES_PerformSearch1984<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1985: CUDA_DES_PerformSearch1985<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1986: CUDA_DES_PerformSearch1986<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1987: CUDA_DES_PerformSearch1987<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1988: CUDA_DES_PerformSearch1988<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1989: CUDA_DES_PerformSearch1989<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1990: CUDA_DES_PerformSearch1990<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1991: CUDA_DES_PerformSearch1991<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1992: CUDA_DES_PerformSearch1992<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1993: CUDA_DES_PerformSearch1993<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1994: CUDA_DES_PerformSearch1994<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1995: CUDA_DES_PerformSearch1995<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1996: CUDA_DES_PerformSearch1996<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1997: CUDA_DES_PerformSearch1997<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1998: CUDA_DES_PerformSearch1998<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1999: CUDA_DES_PerformSearch1999<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2000: CUDA_DES_PerformSearch2000<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2001: CUDA_DES_PerformSearch2001<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2002: CUDA_DES_PerformSearch2002<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2003: CUDA_DES_PerformSearch2003<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2004: CUDA_DES_PerformSearch2004<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2005: CUDA_DES_PerformSearch2005<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2006: CUDA_DES_PerformSearch2006<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2007: CUDA_DES_PerformSearch2007<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2008: CUDA_DES_PerformSearch2008<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2009: CUDA_DES_PerformSearch2009<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2010: CUDA_DES_PerformSearch2010<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2011: CUDA_DES_PerformSearch2011<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2012: CUDA_DES_PerformSearch2012<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2013: CUDA_DES_PerformSearch2013<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2014: CUDA_DES_PerformSearch2014<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2015: CUDA_DES_PerformSearch2015<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2016: CUDA_DES_PerformSearch2016<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2017: CUDA_DES_PerformSearch2017<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2018: CUDA_DES_PerformSearch2018<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2019: CUDA_DES_PerformSearch2019<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2020: CUDA_DES_PerformSearch2020<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2021: CUDA_DES_PerformSearch2021<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2022: CUDA_DES_PerformSearch2022<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2023: CUDA_DES_PerformSearch2023<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2024: CUDA_DES_PerformSearch2024<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2025: CUDA_DES_PerformSearch2025<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2026: CUDA_DES_PerformSearch2026<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2027: CUDA_DES_PerformSearch2027<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2028: CUDA_DES_PerformSearch2028<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2029: CUDA_DES_PerformSearch2029<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2030: CUDA_DES_PerformSearch2030<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2031: CUDA_DES_PerformSearch2031<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2032: CUDA_DES_PerformSearch2032<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2033: CUDA_DES_PerformSearch2033<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2034: CUDA_DES_PerformSearch2034<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2035: CUDA_DES_PerformSearch2035<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2036: CUDA_DES_PerformSearch2036<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2037: CUDA_DES_PerformSearch2037<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2038: CUDA_DES_PerformSearch2038<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2039: CUDA_DES_PerformSearch2039<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2040: CUDA_DES_PerformSearch2040<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2041: CUDA_DES_PerformSearch2041<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2042: CUDA_DES_PerformSearch2042<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2043: CUDA_DES_PerformSearch2043<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2044: CUDA_DES_PerformSearch2044<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2045: CUDA_DES_PerformSearch2045<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2046: CUDA_DES_PerformSearch2046<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2047: CUDA_DES_PerformSearch2047<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
