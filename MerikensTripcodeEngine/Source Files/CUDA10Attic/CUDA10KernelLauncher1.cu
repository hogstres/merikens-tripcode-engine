// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 512
#include "CUDA10Kernel.h"
#define SALT 513
#include "CUDA10Kernel.h"
#define SALT 514
#include "CUDA10Kernel.h"
#define SALT 515
#include "CUDA10Kernel.h"
#define SALT 516
#include "CUDA10Kernel.h"
#define SALT 517
#include "CUDA10Kernel.h"
#define SALT 518
#include "CUDA10Kernel.h"
#define SALT 519
#include "CUDA10Kernel.h"
#define SALT 520
#include "CUDA10Kernel.h"
#define SALT 521
#include "CUDA10Kernel.h"
#define SALT 522
#include "CUDA10Kernel.h"
#define SALT 523
#include "CUDA10Kernel.h"
#define SALT 524
#include "CUDA10Kernel.h"
#define SALT 525
#include "CUDA10Kernel.h"
#define SALT 526
#include "CUDA10Kernel.h"
#define SALT 527
#include "CUDA10Kernel.h"
#define SALT 528
#include "CUDA10Kernel.h"
#define SALT 529
#include "CUDA10Kernel.h"
#define SALT 530
#include "CUDA10Kernel.h"
#define SALT 531
#include "CUDA10Kernel.h"
#define SALT 532
#include "CUDA10Kernel.h"
#define SALT 533
#include "CUDA10Kernel.h"
#define SALT 534
#include "CUDA10Kernel.h"
#define SALT 535
#include "CUDA10Kernel.h"
#define SALT 536
#include "CUDA10Kernel.h"
#define SALT 537
#include "CUDA10Kernel.h"
#define SALT 538
#include "CUDA10Kernel.h"
#define SALT 539
#include "CUDA10Kernel.h"
#define SALT 540
#include "CUDA10Kernel.h"
#define SALT 541
#include "CUDA10Kernel.h"
#define SALT 542
#include "CUDA10Kernel.h"
#define SALT 543
#include "CUDA10Kernel.h"
#define SALT 544
#include "CUDA10Kernel.h"
#define SALT 545
#include "CUDA10Kernel.h"
#define SALT 546
#include "CUDA10Kernel.h"
#define SALT 547
#include "CUDA10Kernel.h"
#define SALT 548
#include "CUDA10Kernel.h"
#define SALT 549
#include "CUDA10Kernel.h"
#define SALT 550
#include "CUDA10Kernel.h"
#define SALT 551
#include "CUDA10Kernel.h"
#define SALT 552
#include "CUDA10Kernel.h"
#define SALT 553
#include "CUDA10Kernel.h"
#define SALT 554
#include "CUDA10Kernel.h"
#define SALT 555
#include "CUDA10Kernel.h"
#define SALT 556
#include "CUDA10Kernel.h"
#define SALT 557
#include "CUDA10Kernel.h"
#define SALT 558
#include "CUDA10Kernel.h"
#define SALT 559
#include "CUDA10Kernel.h"
#define SALT 560
#include "CUDA10Kernel.h"
#define SALT 561
#include "CUDA10Kernel.h"
#define SALT 562
#include "CUDA10Kernel.h"
#define SALT 563
#include "CUDA10Kernel.h"
#define SALT 564
#include "CUDA10Kernel.h"
#define SALT 565
#include "CUDA10Kernel.h"
#define SALT 566
#include "CUDA10Kernel.h"
#define SALT 567
#include "CUDA10Kernel.h"
#define SALT 568
#include "CUDA10Kernel.h"
#define SALT 569
#include "CUDA10Kernel.h"
#define SALT 570
#include "CUDA10Kernel.h"
#define SALT 571
#include "CUDA10Kernel.h"
#define SALT 572
#include "CUDA10Kernel.h"
#define SALT 573
#include "CUDA10Kernel.h"
#define SALT 574
#include "CUDA10Kernel.h"
#define SALT 575
#include "CUDA10Kernel.h"
#define SALT 576
#include "CUDA10Kernel.h"
#define SALT 577
#include "CUDA10Kernel.h"
#define SALT 578
#include "CUDA10Kernel.h"
#define SALT 579
#include "CUDA10Kernel.h"
#define SALT 580
#include "CUDA10Kernel.h"
#define SALT 581
#include "CUDA10Kernel.h"
#define SALT 582
#include "CUDA10Kernel.h"
#define SALT 583
#include "CUDA10Kernel.h"
#define SALT 584
#include "CUDA10Kernel.h"
#define SALT 585
#include "CUDA10Kernel.h"
#define SALT 586
#include "CUDA10Kernel.h"
#define SALT 587
#include "CUDA10Kernel.h"
#define SALT 588
#include "CUDA10Kernel.h"
#define SALT 589
#include "CUDA10Kernel.h"
#define SALT 590
#include "CUDA10Kernel.h"
#define SALT 591
#include "CUDA10Kernel.h"
#define SALT 592
#include "CUDA10Kernel.h"
#define SALT 593
#include "CUDA10Kernel.h"
#define SALT 594
#include "CUDA10Kernel.h"
#define SALT 595
#include "CUDA10Kernel.h"
#define SALT 596
#include "CUDA10Kernel.h"
#define SALT 597
#include "CUDA10Kernel.h"
#define SALT 598
#include "CUDA10Kernel.h"
#define SALT 599
#include "CUDA10Kernel.h"
#define SALT 600
#include "CUDA10Kernel.h"
#define SALT 601
#include "CUDA10Kernel.h"
#define SALT 602
#include "CUDA10Kernel.h"
#define SALT 603
#include "CUDA10Kernel.h"
#define SALT 604
#include "CUDA10Kernel.h"
#define SALT 605
#include "CUDA10Kernel.h"
#define SALT 606
#include "CUDA10Kernel.h"
#define SALT 607
#include "CUDA10Kernel.h"
#define SALT 608
#include "CUDA10Kernel.h"
#define SALT 609
#include "CUDA10Kernel.h"
#define SALT 610
#include "CUDA10Kernel.h"
#define SALT 611
#include "CUDA10Kernel.h"
#define SALT 612
#include "CUDA10Kernel.h"
#define SALT 613
#include "CUDA10Kernel.h"
#define SALT 614
#include "CUDA10Kernel.h"
#define SALT 615
#include "CUDA10Kernel.h"
#define SALT 616
#include "CUDA10Kernel.h"
#define SALT 617
#include "CUDA10Kernel.h"
#define SALT 618
#include "CUDA10Kernel.h"
#define SALT 619
#include "CUDA10Kernel.h"
#define SALT 620
#include "CUDA10Kernel.h"
#define SALT 621
#include "CUDA10Kernel.h"
#define SALT 622
#include "CUDA10Kernel.h"
#define SALT 623
#include "CUDA10Kernel.h"
#define SALT 624
#include "CUDA10Kernel.h"
#define SALT 625
#include "CUDA10Kernel.h"
#define SALT 626
#include "CUDA10Kernel.h"
#define SALT 627
#include "CUDA10Kernel.h"
#define SALT 628
#include "CUDA10Kernel.h"
#define SALT 629
#include "CUDA10Kernel.h"
#define SALT 630
#include "CUDA10Kernel.h"
#define SALT 631
#include "CUDA10Kernel.h"
#define SALT 632
#include "CUDA10Kernel.h"
#define SALT 633
#include "CUDA10Kernel.h"
#define SALT 634
#include "CUDA10Kernel.h"
#define SALT 635
#include "CUDA10Kernel.h"
#define SALT 636
#include "CUDA10Kernel.h"
#define SALT 637
#include "CUDA10Kernel.h"
#define SALT 638
#include "CUDA10Kernel.h"
#define SALT 639
#include "CUDA10Kernel.h"
#define SALT 640
#include "CUDA10Kernel.h"
#define SALT 641
#include "CUDA10Kernel.h"
#define SALT 642
#include "CUDA10Kernel.h"
#define SALT 643
#include "CUDA10Kernel.h"
#define SALT 644
#include "CUDA10Kernel.h"
#define SALT 645
#include "CUDA10Kernel.h"
#define SALT 646
#include "CUDA10Kernel.h"
#define SALT 647
#include "CUDA10Kernel.h"
#define SALT 648
#include "CUDA10Kernel.h"
#define SALT 649
#include "CUDA10Kernel.h"
#define SALT 650
#include "CUDA10Kernel.h"
#define SALT 651
#include "CUDA10Kernel.h"
#define SALT 652
#include "CUDA10Kernel.h"
#define SALT 653
#include "CUDA10Kernel.h"
#define SALT 654
#include "CUDA10Kernel.h"
#define SALT 655
#include "CUDA10Kernel.h"
#define SALT 656
#include "CUDA10Kernel.h"
#define SALT 657
#include "CUDA10Kernel.h"
#define SALT 658
#include "CUDA10Kernel.h"
#define SALT 659
#include "CUDA10Kernel.h"
#define SALT 660
#include "CUDA10Kernel.h"
#define SALT 661
#include "CUDA10Kernel.h"
#define SALT 662
#include "CUDA10Kernel.h"
#define SALT 663
#include "CUDA10Kernel.h"
#define SALT 664
#include "CUDA10Kernel.h"
#define SALT 665
#include "CUDA10Kernel.h"
#define SALT 666
#include "CUDA10Kernel.h"
#define SALT 667
#include "CUDA10Kernel.h"
#define SALT 668
#include "CUDA10Kernel.h"
#define SALT 669
#include "CUDA10Kernel.h"
#define SALT 670
#include "CUDA10Kernel.h"
#define SALT 671
#include "CUDA10Kernel.h"
#define SALT 672
#include "CUDA10Kernel.h"
#define SALT 673
#include "CUDA10Kernel.h"
#define SALT 674
#include "CUDA10Kernel.h"
#define SALT 675
#include "CUDA10Kernel.h"
#define SALT 676
#include "CUDA10Kernel.h"
#define SALT 677
#include "CUDA10Kernel.h"
#define SALT 678
#include "CUDA10Kernel.h"
#define SALT 679
#include "CUDA10Kernel.h"
#define SALT 680
#include "CUDA10Kernel.h"
#define SALT 681
#include "CUDA10Kernel.h"
#define SALT 682
#include "CUDA10Kernel.h"
#define SALT 683
#include "CUDA10Kernel.h"
#define SALT 684
#include "CUDA10Kernel.h"
#define SALT 685
#include "CUDA10Kernel.h"
#define SALT 686
#include "CUDA10Kernel.h"
#define SALT 687
#include "CUDA10Kernel.h"
#define SALT 688
#include "CUDA10Kernel.h"
#define SALT 689
#include "CUDA10Kernel.h"
#define SALT 690
#include "CUDA10Kernel.h"
#define SALT 691
#include "CUDA10Kernel.h"
#define SALT 692
#include "CUDA10Kernel.h"
#define SALT 693
#include "CUDA10Kernel.h"
#define SALT 694
#include "CUDA10Kernel.h"
#define SALT 695
#include "CUDA10Kernel.h"
#define SALT 696
#include "CUDA10Kernel.h"
#define SALT 697
#include "CUDA10Kernel.h"
#define SALT 698
#include "CUDA10Kernel.h"
#define SALT 699
#include "CUDA10Kernel.h"
#define SALT 700
#include "CUDA10Kernel.h"
#define SALT 701
#include "CUDA10Kernel.h"
#define SALT 702
#include "CUDA10Kernel.h"
#define SALT 703
#include "CUDA10Kernel.h"
#define SALT 704
#include "CUDA10Kernel.h"
#define SALT 705
#include "CUDA10Kernel.h"
#define SALT 706
#include "CUDA10Kernel.h"
#define SALT 707
#include "CUDA10Kernel.h"
#define SALT 708
#include "CUDA10Kernel.h"
#define SALT 709
#include "CUDA10Kernel.h"
#define SALT 710
#include "CUDA10Kernel.h"
#define SALT 711
#include "CUDA10Kernel.h"
#define SALT 712
#include "CUDA10Kernel.h"
#define SALT 713
#include "CUDA10Kernel.h"
#define SALT 714
#include "CUDA10Kernel.h"
#define SALT 715
#include "CUDA10Kernel.h"
#define SALT 716
#include "CUDA10Kernel.h"
#define SALT 717
#include "CUDA10Kernel.h"
#define SALT 718
#include "CUDA10Kernel.h"
#define SALT 719
#include "CUDA10Kernel.h"
#define SALT 720
#include "CUDA10Kernel.h"
#define SALT 721
#include "CUDA10Kernel.h"
#define SALT 722
#include "CUDA10Kernel.h"
#define SALT 723
#include "CUDA10Kernel.h"
#define SALT 724
#include "CUDA10Kernel.h"
#define SALT 725
#include "CUDA10Kernel.h"
#define SALT 726
#include "CUDA10Kernel.h"
#define SALT 727
#include "CUDA10Kernel.h"
#define SALT 728
#include "CUDA10Kernel.h"
#define SALT 729
#include "CUDA10Kernel.h"
#define SALT 730
#include "CUDA10Kernel.h"
#define SALT 731
#include "CUDA10Kernel.h"
#define SALT 732
#include "CUDA10Kernel.h"
#define SALT 733
#include "CUDA10Kernel.h"
#define SALT 734
#include "CUDA10Kernel.h"
#define SALT 735
#include "CUDA10Kernel.h"
#define SALT 736
#include "CUDA10Kernel.h"
#define SALT 737
#include "CUDA10Kernel.h"
#define SALT 738
#include "CUDA10Kernel.h"
#define SALT 739
#include "CUDA10Kernel.h"
#define SALT 740
#include "CUDA10Kernel.h"
#define SALT 741
#include "CUDA10Kernel.h"
#define SALT 742
#include "CUDA10Kernel.h"
#define SALT 743
#include "CUDA10Kernel.h"
#define SALT 744
#include "CUDA10Kernel.h"
#define SALT 745
#include "CUDA10Kernel.h"
#define SALT 746
#include "CUDA10Kernel.h"
#define SALT 747
#include "CUDA10Kernel.h"
#define SALT 748
#include "CUDA10Kernel.h"
#define SALT 749
#include "CUDA10Kernel.h"
#define SALT 750
#include "CUDA10Kernel.h"
#define SALT 751
#include "CUDA10Kernel.h"
#define SALT 752
#include "CUDA10Kernel.h"
#define SALT 753
#include "CUDA10Kernel.h"
#define SALT 754
#include "CUDA10Kernel.h"
#define SALT 755
#include "CUDA10Kernel.h"
#define SALT 756
#include "CUDA10Kernel.h"
#define SALT 757
#include "CUDA10Kernel.h"
#define SALT 758
#include "CUDA10Kernel.h"
#define SALT 759
#include "CUDA10Kernel.h"
#define SALT 760
#include "CUDA10Kernel.h"
#define SALT 761
#include "CUDA10Kernel.h"
#define SALT 762
#include "CUDA10Kernel.h"
#define SALT 763
#include "CUDA10Kernel.h"
#define SALT 764
#include "CUDA10Kernel.h"
#define SALT 765
#include "CUDA10Kernel.h"
#define SALT 766
#include "CUDA10Kernel.h"
#define SALT 767
#include "CUDA10Kernel.h"
#define SALT 768
#include "CUDA10Kernel.h"
#define SALT 769
#include "CUDA10Kernel.h"
#define SALT 770
#include "CUDA10Kernel.h"
#define SALT 771
#include "CUDA10Kernel.h"
#define SALT 772
#include "CUDA10Kernel.h"
#define SALT 773
#include "CUDA10Kernel.h"
#define SALT 774
#include "CUDA10Kernel.h"
#define SALT 775
#include "CUDA10Kernel.h"
#define SALT 776
#include "CUDA10Kernel.h"
#define SALT 777
#include "CUDA10Kernel.h"
#define SALT 778
#include "CUDA10Kernel.h"
#define SALT 779
#include "CUDA10Kernel.h"
#define SALT 780
#include "CUDA10Kernel.h"
#define SALT 781
#include "CUDA10Kernel.h"
#define SALT 782
#include "CUDA10Kernel.h"
#define SALT 783
#include "CUDA10Kernel.h"
#define SALT 784
#include "CUDA10Kernel.h"
#define SALT 785
#include "CUDA10Kernel.h"
#define SALT 786
#include "CUDA10Kernel.h"
#define SALT 787
#include "CUDA10Kernel.h"
#define SALT 788
#include "CUDA10Kernel.h"
#define SALT 789
#include "CUDA10Kernel.h"
#define SALT 790
#include "CUDA10Kernel.h"
#define SALT 791
#include "CUDA10Kernel.h"
#define SALT 792
#include "CUDA10Kernel.h"
#define SALT 793
#include "CUDA10Kernel.h"
#define SALT 794
#include "CUDA10Kernel.h"
#define SALT 795
#include "CUDA10Kernel.h"
#define SALT 796
#include "CUDA10Kernel.h"
#define SALT 797
#include "CUDA10Kernel.h"
#define SALT 798
#include "CUDA10Kernel.h"
#define SALT 799
#include "CUDA10Kernel.h"
#define SALT 800
#include "CUDA10Kernel.h"
#define SALT 801
#include "CUDA10Kernel.h"
#define SALT 802
#include "CUDA10Kernel.h"
#define SALT 803
#include "CUDA10Kernel.h"
#define SALT 804
#include "CUDA10Kernel.h"
#define SALT 805
#include "CUDA10Kernel.h"
#define SALT 806
#include "CUDA10Kernel.h"
#define SALT 807
#include "CUDA10Kernel.h"
#define SALT 808
#include "CUDA10Kernel.h"
#define SALT 809
#include "CUDA10Kernel.h"
#define SALT 810
#include "CUDA10Kernel.h"
#define SALT 811
#include "CUDA10Kernel.h"
#define SALT 812
#include "CUDA10Kernel.h"
#define SALT 813
#include "CUDA10Kernel.h"
#define SALT 814
#include "CUDA10Kernel.h"
#define SALT 815
#include "CUDA10Kernel.h"
#define SALT 816
#include "CUDA10Kernel.h"
#define SALT 817
#include "CUDA10Kernel.h"
#define SALT 818
#include "CUDA10Kernel.h"
#define SALT 819
#include "CUDA10Kernel.h"
#define SALT 820
#include "CUDA10Kernel.h"
#define SALT 821
#include "CUDA10Kernel.h"
#define SALT 822
#include "CUDA10Kernel.h"
#define SALT 823
#include "CUDA10Kernel.h"
#define SALT 824
#include "CUDA10Kernel.h"
#define SALT 825
#include "CUDA10Kernel.h"
#define SALT 826
#include "CUDA10Kernel.h"
#define SALT 827
#include "CUDA10Kernel.h"
#define SALT 828
#include "CUDA10Kernel.h"
#define SALT 829
#include "CUDA10Kernel.h"
#define SALT 830
#include "CUDA10Kernel.h"
#define SALT 831
#include "CUDA10Kernel.h"
#define SALT 832
#include "CUDA10Kernel.h"
#define SALT 833
#include "CUDA10Kernel.h"
#define SALT 834
#include "CUDA10Kernel.h"
#define SALT 835
#include "CUDA10Kernel.h"
#define SALT 836
#include "CUDA10Kernel.h"
#define SALT 837
#include "CUDA10Kernel.h"
#define SALT 838
#include "CUDA10Kernel.h"
#define SALT 839
#include "CUDA10Kernel.h"
#define SALT 840
#include "CUDA10Kernel.h"
#define SALT 841
#include "CUDA10Kernel.h"
#define SALT 842
#include "CUDA10Kernel.h"
#define SALT 843
#include "CUDA10Kernel.h"
#define SALT 844
#include "CUDA10Kernel.h"
#define SALT 845
#include "CUDA10Kernel.h"
#define SALT 846
#include "CUDA10Kernel.h"
#define SALT 847
#include "CUDA10Kernel.h"
#define SALT 848
#include "CUDA10Kernel.h"
#define SALT 849
#include "CUDA10Kernel.h"
#define SALT 850
#include "CUDA10Kernel.h"
#define SALT 851
#include "CUDA10Kernel.h"
#define SALT 852
#include "CUDA10Kernel.h"
#define SALT 853
#include "CUDA10Kernel.h"
#define SALT 854
#include "CUDA10Kernel.h"
#define SALT 855
#include "CUDA10Kernel.h"
#define SALT 856
#include "CUDA10Kernel.h"
#define SALT 857
#include "CUDA10Kernel.h"
#define SALT 858
#include "CUDA10Kernel.h"
#define SALT 859
#include "CUDA10Kernel.h"
#define SALT 860
#include "CUDA10Kernel.h"
#define SALT 861
#include "CUDA10Kernel.h"
#define SALT 862
#include "CUDA10Kernel.h"
#define SALT 863
#include "CUDA10Kernel.h"
#define SALT 864
#include "CUDA10Kernel.h"
#define SALT 865
#include "CUDA10Kernel.h"
#define SALT 866
#include "CUDA10Kernel.h"
#define SALT 867
#include "CUDA10Kernel.h"
#define SALT 868
#include "CUDA10Kernel.h"
#define SALT 869
#include "CUDA10Kernel.h"
#define SALT 870
#include "CUDA10Kernel.h"
#define SALT 871
#include "CUDA10Kernel.h"
#define SALT 872
#include "CUDA10Kernel.h"
#define SALT 873
#include "CUDA10Kernel.h"
#define SALT 874
#include "CUDA10Kernel.h"
#define SALT 875
#include "CUDA10Kernel.h"
#define SALT 876
#include "CUDA10Kernel.h"
#define SALT 877
#include "CUDA10Kernel.h"
#define SALT 878
#include "CUDA10Kernel.h"
#define SALT 879
#include "CUDA10Kernel.h"
#define SALT 880
#include "CUDA10Kernel.h"
#define SALT 881
#include "CUDA10Kernel.h"
#define SALT 882
#include "CUDA10Kernel.h"
#define SALT 883
#include "CUDA10Kernel.h"
#define SALT 884
#include "CUDA10Kernel.h"
#define SALT 885
#include "CUDA10Kernel.h"
#define SALT 886
#include "CUDA10Kernel.h"
#define SALT 887
#include "CUDA10Kernel.h"
#define SALT 888
#include "CUDA10Kernel.h"
#define SALT 889
#include "CUDA10Kernel.h"
#define SALT 890
#include "CUDA10Kernel.h"
#define SALT 891
#include "CUDA10Kernel.h"
#define SALT 892
#include "CUDA10Kernel.h"
#define SALT 893
#include "CUDA10Kernel.h"
#define SALT 894
#include "CUDA10Kernel.h"
#define SALT 895
#include "CUDA10Kernel.h"
#define SALT 896
#include "CUDA10Kernel.h"
#define SALT 897
#include "CUDA10Kernel.h"
#define SALT 898
#include "CUDA10Kernel.h"
#define SALT 899
#include "CUDA10Kernel.h"
#define SALT 900
#include "CUDA10Kernel.h"
#define SALT 901
#include "CUDA10Kernel.h"
#define SALT 902
#include "CUDA10Kernel.h"
#define SALT 903
#include "CUDA10Kernel.h"
#define SALT 904
#include "CUDA10Kernel.h"
#define SALT 905
#include "CUDA10Kernel.h"
#define SALT 906
#include "CUDA10Kernel.h"
#define SALT 907
#include "CUDA10Kernel.h"
#define SALT 908
#include "CUDA10Kernel.h"
#define SALT 909
#include "CUDA10Kernel.h"
#define SALT 910
#include "CUDA10Kernel.h"
#define SALT 911
#include "CUDA10Kernel.h"
#define SALT 912
#include "CUDA10Kernel.h"
#define SALT 913
#include "CUDA10Kernel.h"
#define SALT 914
#include "CUDA10Kernel.h"
#define SALT 915
#include "CUDA10Kernel.h"
#define SALT 916
#include "CUDA10Kernel.h"
#define SALT 917
#include "CUDA10Kernel.h"
#define SALT 918
#include "CUDA10Kernel.h"
#define SALT 919
#include "CUDA10Kernel.h"
#define SALT 920
#include "CUDA10Kernel.h"
#define SALT 921
#include "CUDA10Kernel.h"
#define SALT 922
#include "CUDA10Kernel.h"
#define SALT 923
#include "CUDA10Kernel.h"
#define SALT 924
#include "CUDA10Kernel.h"
#define SALT 925
#include "CUDA10Kernel.h"
#define SALT 926
#include "CUDA10Kernel.h"
#define SALT 927
#include "CUDA10Kernel.h"
#define SALT 928
#include "CUDA10Kernel.h"
#define SALT 929
#include "CUDA10Kernel.h"
#define SALT 930
#include "CUDA10Kernel.h"
#define SALT 931
#include "CUDA10Kernel.h"
#define SALT 932
#include "CUDA10Kernel.h"
#define SALT 933
#include "CUDA10Kernel.h"
#define SALT 934
#include "CUDA10Kernel.h"
#define SALT 935
#include "CUDA10Kernel.h"
#define SALT 936
#include "CUDA10Kernel.h"
#define SALT 937
#include "CUDA10Kernel.h"
#define SALT 938
#include "CUDA10Kernel.h"
#define SALT 939
#include "CUDA10Kernel.h"
#define SALT 940
#include "CUDA10Kernel.h"
#define SALT 941
#include "CUDA10Kernel.h"
#define SALT 942
#include "CUDA10Kernel.h"
#define SALT 943
#include "CUDA10Kernel.h"
#define SALT 944
#include "CUDA10Kernel.h"
#define SALT 945
#include "CUDA10Kernel.h"
#define SALT 946
#include "CUDA10Kernel.h"
#define SALT 947
#include "CUDA10Kernel.h"
#define SALT 948
#include "CUDA10Kernel.h"
#define SALT 949
#include "CUDA10Kernel.h"
#define SALT 950
#include "CUDA10Kernel.h"
#define SALT 951
#include "CUDA10Kernel.h"
#define SALT 952
#include "CUDA10Kernel.h"
#define SALT 953
#include "CUDA10Kernel.h"
#define SALT 954
#include "CUDA10Kernel.h"
#define SALT 955
#include "CUDA10Kernel.h"
#define SALT 956
#include "CUDA10Kernel.h"
#define SALT 957
#include "CUDA10Kernel.h"
#define SALT 958
#include "CUDA10Kernel.h"
#define SALT 959
#include "CUDA10Kernel.h"
#define SALT 960
#include "CUDA10Kernel.h"
#define SALT 961
#include "CUDA10Kernel.h"
#define SALT 962
#include "CUDA10Kernel.h"
#define SALT 963
#include "CUDA10Kernel.h"
#define SALT 964
#include "CUDA10Kernel.h"
#define SALT 965
#include "CUDA10Kernel.h"
#define SALT 966
#include "CUDA10Kernel.h"
#define SALT 967
#include "CUDA10Kernel.h"
#define SALT 968
#include "CUDA10Kernel.h"
#define SALT 969
#include "CUDA10Kernel.h"
#define SALT 970
#include "CUDA10Kernel.h"
#define SALT 971
#include "CUDA10Kernel.h"
#define SALT 972
#include "CUDA10Kernel.h"
#define SALT 973
#include "CUDA10Kernel.h"
#define SALT 974
#include "CUDA10Kernel.h"
#define SALT 975
#include "CUDA10Kernel.h"
#define SALT 976
#include "CUDA10Kernel.h"
#define SALT 977
#include "CUDA10Kernel.h"
#define SALT 978
#include "CUDA10Kernel.h"
#define SALT 979
#include "CUDA10Kernel.h"
#define SALT 980
#include "CUDA10Kernel.h"
#define SALT 981
#include "CUDA10Kernel.h"
#define SALT 982
#include "CUDA10Kernel.h"
#define SALT 983
#include "CUDA10Kernel.h"
#define SALT 984
#include "CUDA10Kernel.h"
#define SALT 985
#include "CUDA10Kernel.h"
#define SALT 986
#include "CUDA10Kernel.h"
#define SALT 987
#include "CUDA10Kernel.h"
#define SALT 988
#include "CUDA10Kernel.h"
#define SALT 989
#include "CUDA10Kernel.h"
#define SALT 990
#include "CUDA10Kernel.h"
#define SALT 991
#include "CUDA10Kernel.h"
#define SALT 992
#include "CUDA10Kernel.h"
#define SALT 993
#include "CUDA10Kernel.h"
#define SALT 994
#include "CUDA10Kernel.h"
#define SALT 995
#include "CUDA10Kernel.h"
#define SALT 996
#include "CUDA10Kernel.h"
#define SALT 997
#include "CUDA10Kernel.h"
#define SALT 998
#include "CUDA10Kernel.h"
#define SALT 999
#include "CUDA10Kernel.h"
#define SALT 1000
#include "CUDA10Kernel.h"
#define SALT 1001
#include "CUDA10Kernel.h"
#define SALT 1002
#include "CUDA10Kernel.h"
#define SALT 1003
#include "CUDA10Kernel.h"
#define SALT 1004
#include "CUDA10Kernel.h"
#define SALT 1005
#include "CUDA10Kernel.h"
#define SALT 1006
#include "CUDA10Kernel.h"
#define SALT 1007
#include "CUDA10Kernel.h"
#define SALT 1008
#include "CUDA10Kernel.h"
#define SALT 1009
#include "CUDA10Kernel.h"
#define SALT 1010
#include "CUDA10Kernel.h"
#define SALT 1011
#include "CUDA10Kernel.h"
#define SALT 1012
#include "CUDA10Kernel.h"
#define SALT 1013
#include "CUDA10Kernel.h"
#define SALT 1014
#include "CUDA10Kernel.h"
#define SALT 1015
#include "CUDA10Kernel.h"
#define SALT 1016
#include "CUDA10Kernel.h"
#define SALT 1017
#include "CUDA10Kernel.h"
#define SALT 1018
#include "CUDA10Kernel.h"
#define SALT 1019
#include "CUDA10Kernel.h"
#define SALT 1020
#include "CUDA10Kernel.h"
#define SALT 1021
#include "CUDA10Kernel.h"
#define SALT 1022
#include "CUDA10Kernel.h"
#define SALT 1023
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher1()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel1(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 512: CUDA_DES_PerformSearch512<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 513: CUDA_DES_PerformSearch513<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 514: CUDA_DES_PerformSearch514<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 515: CUDA_DES_PerformSearch515<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 516: CUDA_DES_PerformSearch516<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 517: CUDA_DES_PerformSearch517<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 518: CUDA_DES_PerformSearch518<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 519: CUDA_DES_PerformSearch519<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 520: CUDA_DES_PerformSearch520<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 521: CUDA_DES_PerformSearch521<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 522: CUDA_DES_PerformSearch522<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 523: CUDA_DES_PerformSearch523<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 524: CUDA_DES_PerformSearch524<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 525: CUDA_DES_PerformSearch525<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 526: CUDA_DES_PerformSearch526<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 527: CUDA_DES_PerformSearch527<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 528: CUDA_DES_PerformSearch528<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 529: CUDA_DES_PerformSearch529<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 530: CUDA_DES_PerformSearch530<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 531: CUDA_DES_PerformSearch531<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 532: CUDA_DES_PerformSearch532<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 533: CUDA_DES_PerformSearch533<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 534: CUDA_DES_PerformSearch534<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 535: CUDA_DES_PerformSearch535<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 536: CUDA_DES_PerformSearch536<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 537: CUDA_DES_PerformSearch537<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 538: CUDA_DES_PerformSearch538<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 539: CUDA_DES_PerformSearch539<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 540: CUDA_DES_PerformSearch540<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 541: CUDA_DES_PerformSearch541<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 542: CUDA_DES_PerformSearch542<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 543: CUDA_DES_PerformSearch543<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 544: CUDA_DES_PerformSearch544<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 545: CUDA_DES_PerformSearch545<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 546: CUDA_DES_PerformSearch546<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 547: CUDA_DES_PerformSearch547<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 548: CUDA_DES_PerformSearch548<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 549: CUDA_DES_PerformSearch549<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 550: CUDA_DES_PerformSearch550<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 551: CUDA_DES_PerformSearch551<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 552: CUDA_DES_PerformSearch552<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 553: CUDA_DES_PerformSearch553<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 554: CUDA_DES_PerformSearch554<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 555: CUDA_DES_PerformSearch555<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 556: CUDA_DES_PerformSearch556<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 557: CUDA_DES_PerformSearch557<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 558: CUDA_DES_PerformSearch558<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 559: CUDA_DES_PerformSearch559<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 560: CUDA_DES_PerformSearch560<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 561: CUDA_DES_PerformSearch561<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 562: CUDA_DES_PerformSearch562<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 563: CUDA_DES_PerformSearch563<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 564: CUDA_DES_PerformSearch564<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 565: CUDA_DES_PerformSearch565<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 566: CUDA_DES_PerformSearch566<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 567: CUDA_DES_PerformSearch567<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 568: CUDA_DES_PerformSearch568<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 569: CUDA_DES_PerformSearch569<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 570: CUDA_DES_PerformSearch570<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 571: CUDA_DES_PerformSearch571<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 572: CUDA_DES_PerformSearch572<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 573: CUDA_DES_PerformSearch573<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 574: CUDA_DES_PerformSearch574<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 575: CUDA_DES_PerformSearch575<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 576: CUDA_DES_PerformSearch576<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 577: CUDA_DES_PerformSearch577<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 578: CUDA_DES_PerformSearch578<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 579: CUDA_DES_PerformSearch579<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 580: CUDA_DES_PerformSearch580<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 581: CUDA_DES_PerformSearch581<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 582: CUDA_DES_PerformSearch582<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 583: CUDA_DES_PerformSearch583<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 584: CUDA_DES_PerformSearch584<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 585: CUDA_DES_PerformSearch585<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 586: CUDA_DES_PerformSearch586<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 587: CUDA_DES_PerformSearch587<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 588: CUDA_DES_PerformSearch588<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 589: CUDA_DES_PerformSearch589<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 590: CUDA_DES_PerformSearch590<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 591: CUDA_DES_PerformSearch591<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 592: CUDA_DES_PerformSearch592<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 593: CUDA_DES_PerformSearch593<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 594: CUDA_DES_PerformSearch594<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 595: CUDA_DES_PerformSearch595<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 596: CUDA_DES_PerformSearch596<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 597: CUDA_DES_PerformSearch597<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 598: CUDA_DES_PerformSearch598<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 599: CUDA_DES_PerformSearch599<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 600: CUDA_DES_PerformSearch600<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 601: CUDA_DES_PerformSearch601<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 602: CUDA_DES_PerformSearch602<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 603: CUDA_DES_PerformSearch603<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 604: CUDA_DES_PerformSearch604<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 605: CUDA_DES_PerformSearch605<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 606: CUDA_DES_PerformSearch606<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 607: CUDA_DES_PerformSearch607<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 608: CUDA_DES_PerformSearch608<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 609: CUDA_DES_PerformSearch609<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 610: CUDA_DES_PerformSearch610<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 611: CUDA_DES_PerformSearch611<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 612: CUDA_DES_PerformSearch612<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 613: CUDA_DES_PerformSearch613<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 614: CUDA_DES_PerformSearch614<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 615: CUDA_DES_PerformSearch615<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 616: CUDA_DES_PerformSearch616<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 617: CUDA_DES_PerformSearch617<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 618: CUDA_DES_PerformSearch618<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 619: CUDA_DES_PerformSearch619<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 620: CUDA_DES_PerformSearch620<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 621: CUDA_DES_PerformSearch621<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 622: CUDA_DES_PerformSearch622<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 623: CUDA_DES_PerformSearch623<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 624: CUDA_DES_PerformSearch624<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 625: CUDA_DES_PerformSearch625<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 626: CUDA_DES_PerformSearch626<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 627: CUDA_DES_PerformSearch627<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 628: CUDA_DES_PerformSearch628<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 629: CUDA_DES_PerformSearch629<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 630: CUDA_DES_PerformSearch630<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 631: CUDA_DES_PerformSearch631<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 632: CUDA_DES_PerformSearch632<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 633: CUDA_DES_PerformSearch633<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 634: CUDA_DES_PerformSearch634<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 635: CUDA_DES_PerformSearch635<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 636: CUDA_DES_PerformSearch636<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 637: CUDA_DES_PerformSearch637<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 638: CUDA_DES_PerformSearch638<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 639: CUDA_DES_PerformSearch639<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 640: CUDA_DES_PerformSearch640<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 641: CUDA_DES_PerformSearch641<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 642: CUDA_DES_PerformSearch642<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 643: CUDA_DES_PerformSearch643<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 644: CUDA_DES_PerformSearch644<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 645: CUDA_DES_PerformSearch645<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 646: CUDA_DES_PerformSearch646<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 647: CUDA_DES_PerformSearch647<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 648: CUDA_DES_PerformSearch648<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 649: CUDA_DES_PerformSearch649<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 650: CUDA_DES_PerformSearch650<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 651: CUDA_DES_PerformSearch651<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 652: CUDA_DES_PerformSearch652<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 653: CUDA_DES_PerformSearch653<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 654: CUDA_DES_PerformSearch654<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 655: CUDA_DES_PerformSearch655<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 656: CUDA_DES_PerformSearch656<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 657: CUDA_DES_PerformSearch657<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 658: CUDA_DES_PerformSearch658<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 659: CUDA_DES_PerformSearch659<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 660: CUDA_DES_PerformSearch660<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 661: CUDA_DES_PerformSearch661<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 662: CUDA_DES_PerformSearch662<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 663: CUDA_DES_PerformSearch663<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 664: CUDA_DES_PerformSearch664<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 665: CUDA_DES_PerformSearch665<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 666: CUDA_DES_PerformSearch666<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 667: CUDA_DES_PerformSearch667<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 668: CUDA_DES_PerformSearch668<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 669: CUDA_DES_PerformSearch669<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 670: CUDA_DES_PerformSearch670<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 671: CUDA_DES_PerformSearch671<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 672: CUDA_DES_PerformSearch672<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 673: CUDA_DES_PerformSearch673<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 674: CUDA_DES_PerformSearch674<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 675: CUDA_DES_PerformSearch675<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 676: CUDA_DES_PerformSearch676<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 677: CUDA_DES_PerformSearch677<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 678: CUDA_DES_PerformSearch678<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 679: CUDA_DES_PerformSearch679<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 680: CUDA_DES_PerformSearch680<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 681: CUDA_DES_PerformSearch681<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 682: CUDA_DES_PerformSearch682<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 683: CUDA_DES_PerformSearch683<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 684: CUDA_DES_PerformSearch684<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 685: CUDA_DES_PerformSearch685<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 686: CUDA_DES_PerformSearch686<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 687: CUDA_DES_PerformSearch687<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 688: CUDA_DES_PerformSearch688<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 689: CUDA_DES_PerformSearch689<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 690: CUDA_DES_PerformSearch690<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 691: CUDA_DES_PerformSearch691<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 692: CUDA_DES_PerformSearch692<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 693: CUDA_DES_PerformSearch693<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 694: CUDA_DES_PerformSearch694<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 695: CUDA_DES_PerformSearch695<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 696: CUDA_DES_PerformSearch696<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 697: CUDA_DES_PerformSearch697<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 698: CUDA_DES_PerformSearch698<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 699: CUDA_DES_PerformSearch699<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 700: CUDA_DES_PerformSearch700<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 701: CUDA_DES_PerformSearch701<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 702: CUDA_DES_PerformSearch702<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 703: CUDA_DES_PerformSearch703<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 704: CUDA_DES_PerformSearch704<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 705: CUDA_DES_PerformSearch705<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 706: CUDA_DES_PerformSearch706<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 707: CUDA_DES_PerformSearch707<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 708: CUDA_DES_PerformSearch708<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 709: CUDA_DES_PerformSearch709<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 710: CUDA_DES_PerformSearch710<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 711: CUDA_DES_PerformSearch711<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 712: CUDA_DES_PerformSearch712<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 713: CUDA_DES_PerformSearch713<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 714: CUDA_DES_PerformSearch714<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 715: CUDA_DES_PerformSearch715<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 716: CUDA_DES_PerformSearch716<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 717: CUDA_DES_PerformSearch717<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 718: CUDA_DES_PerformSearch718<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 719: CUDA_DES_PerformSearch719<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 720: CUDA_DES_PerformSearch720<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 721: CUDA_DES_PerformSearch721<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 722: CUDA_DES_PerformSearch722<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 723: CUDA_DES_PerformSearch723<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 724: CUDA_DES_PerformSearch724<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 725: CUDA_DES_PerformSearch725<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 726: CUDA_DES_PerformSearch726<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 727: CUDA_DES_PerformSearch727<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 728: CUDA_DES_PerformSearch728<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 729: CUDA_DES_PerformSearch729<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 730: CUDA_DES_PerformSearch730<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 731: CUDA_DES_PerformSearch731<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 732: CUDA_DES_PerformSearch732<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 733: CUDA_DES_PerformSearch733<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 734: CUDA_DES_PerformSearch734<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 735: CUDA_DES_PerformSearch735<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 736: CUDA_DES_PerformSearch736<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 737: CUDA_DES_PerformSearch737<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 738: CUDA_DES_PerformSearch738<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 739: CUDA_DES_PerformSearch739<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 740: CUDA_DES_PerformSearch740<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 741: CUDA_DES_PerformSearch741<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 742: CUDA_DES_PerformSearch742<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 743: CUDA_DES_PerformSearch743<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 744: CUDA_DES_PerformSearch744<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 745: CUDA_DES_PerformSearch745<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 746: CUDA_DES_PerformSearch746<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 747: CUDA_DES_PerformSearch747<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 748: CUDA_DES_PerformSearch748<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 749: CUDA_DES_PerformSearch749<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 750: CUDA_DES_PerformSearch750<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 751: CUDA_DES_PerformSearch751<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 752: CUDA_DES_PerformSearch752<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 753: CUDA_DES_PerformSearch753<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 754: CUDA_DES_PerformSearch754<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 755: CUDA_DES_PerformSearch755<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 756: CUDA_DES_PerformSearch756<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 757: CUDA_DES_PerformSearch757<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 758: CUDA_DES_PerformSearch758<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 759: CUDA_DES_PerformSearch759<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 760: CUDA_DES_PerformSearch760<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 761: CUDA_DES_PerformSearch761<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 762: CUDA_DES_PerformSearch762<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 763: CUDA_DES_PerformSearch763<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 764: CUDA_DES_PerformSearch764<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 765: CUDA_DES_PerformSearch765<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 766: CUDA_DES_PerformSearch766<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 767: CUDA_DES_PerformSearch767<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 768: CUDA_DES_PerformSearch768<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 769: CUDA_DES_PerformSearch769<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 770: CUDA_DES_PerformSearch770<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 771: CUDA_DES_PerformSearch771<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 772: CUDA_DES_PerformSearch772<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 773: CUDA_DES_PerformSearch773<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 774: CUDA_DES_PerformSearch774<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 775: CUDA_DES_PerformSearch775<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 776: CUDA_DES_PerformSearch776<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 777: CUDA_DES_PerformSearch777<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 778: CUDA_DES_PerformSearch778<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 779: CUDA_DES_PerformSearch779<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 780: CUDA_DES_PerformSearch780<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 781: CUDA_DES_PerformSearch781<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 782: CUDA_DES_PerformSearch782<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 783: CUDA_DES_PerformSearch783<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 784: CUDA_DES_PerformSearch784<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 785: CUDA_DES_PerformSearch785<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 786: CUDA_DES_PerformSearch786<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 787: CUDA_DES_PerformSearch787<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 788: CUDA_DES_PerformSearch788<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 789: CUDA_DES_PerformSearch789<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 790: CUDA_DES_PerformSearch790<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 791: CUDA_DES_PerformSearch791<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 792: CUDA_DES_PerformSearch792<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 793: CUDA_DES_PerformSearch793<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 794: CUDA_DES_PerformSearch794<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 795: CUDA_DES_PerformSearch795<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 796: CUDA_DES_PerformSearch796<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 797: CUDA_DES_PerformSearch797<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 798: CUDA_DES_PerformSearch798<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 799: CUDA_DES_PerformSearch799<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 800: CUDA_DES_PerformSearch800<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 801: CUDA_DES_PerformSearch801<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 802: CUDA_DES_PerformSearch802<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 803: CUDA_DES_PerformSearch803<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 804: CUDA_DES_PerformSearch804<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 805: CUDA_DES_PerformSearch805<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 806: CUDA_DES_PerformSearch806<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 807: CUDA_DES_PerformSearch807<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 808: CUDA_DES_PerformSearch808<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 809: CUDA_DES_PerformSearch809<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 810: CUDA_DES_PerformSearch810<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 811: CUDA_DES_PerformSearch811<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 812: CUDA_DES_PerformSearch812<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 813: CUDA_DES_PerformSearch813<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 814: CUDA_DES_PerformSearch814<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 815: CUDA_DES_PerformSearch815<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 816: CUDA_DES_PerformSearch816<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 817: CUDA_DES_PerformSearch817<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 818: CUDA_DES_PerformSearch818<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 819: CUDA_DES_PerformSearch819<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 820: CUDA_DES_PerformSearch820<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 821: CUDA_DES_PerformSearch821<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 822: CUDA_DES_PerformSearch822<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 823: CUDA_DES_PerformSearch823<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 824: CUDA_DES_PerformSearch824<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 825: CUDA_DES_PerformSearch825<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 826: CUDA_DES_PerformSearch826<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 827: CUDA_DES_PerformSearch827<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 828: CUDA_DES_PerformSearch828<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 829: CUDA_DES_PerformSearch829<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 830: CUDA_DES_PerformSearch830<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 831: CUDA_DES_PerformSearch831<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 832: CUDA_DES_PerformSearch832<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 833: CUDA_DES_PerformSearch833<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 834: CUDA_DES_PerformSearch834<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 835: CUDA_DES_PerformSearch835<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 836: CUDA_DES_PerformSearch836<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 837: CUDA_DES_PerformSearch837<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 838: CUDA_DES_PerformSearch838<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 839: CUDA_DES_PerformSearch839<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 840: CUDA_DES_PerformSearch840<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 841: CUDA_DES_PerformSearch841<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 842: CUDA_DES_PerformSearch842<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 843: CUDA_DES_PerformSearch843<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 844: CUDA_DES_PerformSearch844<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 845: CUDA_DES_PerformSearch845<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 846: CUDA_DES_PerformSearch846<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 847: CUDA_DES_PerformSearch847<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 848: CUDA_DES_PerformSearch848<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 849: CUDA_DES_PerformSearch849<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 850: CUDA_DES_PerformSearch850<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 851: CUDA_DES_PerformSearch851<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 852: CUDA_DES_PerformSearch852<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 853: CUDA_DES_PerformSearch853<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 854: CUDA_DES_PerformSearch854<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 855: CUDA_DES_PerformSearch855<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 856: CUDA_DES_PerformSearch856<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 857: CUDA_DES_PerformSearch857<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 858: CUDA_DES_PerformSearch858<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 859: CUDA_DES_PerformSearch859<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 860: CUDA_DES_PerformSearch860<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 861: CUDA_DES_PerformSearch861<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 862: CUDA_DES_PerformSearch862<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 863: CUDA_DES_PerformSearch863<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 864: CUDA_DES_PerformSearch864<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 865: CUDA_DES_PerformSearch865<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 866: CUDA_DES_PerformSearch866<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 867: CUDA_DES_PerformSearch867<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 868: CUDA_DES_PerformSearch868<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 869: CUDA_DES_PerformSearch869<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 870: CUDA_DES_PerformSearch870<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 871: CUDA_DES_PerformSearch871<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 872: CUDA_DES_PerformSearch872<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 873: CUDA_DES_PerformSearch873<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 874: CUDA_DES_PerformSearch874<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 875: CUDA_DES_PerformSearch875<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 876: CUDA_DES_PerformSearch876<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 877: CUDA_DES_PerformSearch877<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 878: CUDA_DES_PerformSearch878<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 879: CUDA_DES_PerformSearch879<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 880: CUDA_DES_PerformSearch880<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 881: CUDA_DES_PerformSearch881<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 882: CUDA_DES_PerformSearch882<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 883: CUDA_DES_PerformSearch883<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 884: CUDA_DES_PerformSearch884<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 885: CUDA_DES_PerformSearch885<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 886: CUDA_DES_PerformSearch886<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 887: CUDA_DES_PerformSearch887<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 888: CUDA_DES_PerformSearch888<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 889: CUDA_DES_PerformSearch889<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 890: CUDA_DES_PerformSearch890<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 891: CUDA_DES_PerformSearch891<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 892: CUDA_DES_PerformSearch892<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 893: CUDA_DES_PerformSearch893<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 894: CUDA_DES_PerformSearch894<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 895: CUDA_DES_PerformSearch895<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 896: CUDA_DES_PerformSearch896<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 897: CUDA_DES_PerformSearch897<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 898: CUDA_DES_PerformSearch898<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 899: CUDA_DES_PerformSearch899<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 900: CUDA_DES_PerformSearch900<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 901: CUDA_DES_PerformSearch901<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 902: CUDA_DES_PerformSearch902<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 903: CUDA_DES_PerformSearch903<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 904: CUDA_DES_PerformSearch904<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 905: CUDA_DES_PerformSearch905<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 906: CUDA_DES_PerformSearch906<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 907: CUDA_DES_PerformSearch907<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 908: CUDA_DES_PerformSearch908<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 909: CUDA_DES_PerformSearch909<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 910: CUDA_DES_PerformSearch910<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 911: CUDA_DES_PerformSearch911<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 912: CUDA_DES_PerformSearch912<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 913: CUDA_DES_PerformSearch913<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 914: CUDA_DES_PerformSearch914<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 915: CUDA_DES_PerformSearch915<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 916: CUDA_DES_PerformSearch916<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 917: CUDA_DES_PerformSearch917<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 918: CUDA_DES_PerformSearch918<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 919: CUDA_DES_PerformSearch919<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 920: CUDA_DES_PerformSearch920<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 921: CUDA_DES_PerformSearch921<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 922: CUDA_DES_PerformSearch922<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 923: CUDA_DES_PerformSearch923<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 924: CUDA_DES_PerformSearch924<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 925: CUDA_DES_PerformSearch925<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 926: CUDA_DES_PerformSearch926<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 927: CUDA_DES_PerformSearch927<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 928: CUDA_DES_PerformSearch928<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 929: CUDA_DES_PerformSearch929<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 930: CUDA_DES_PerformSearch930<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 931: CUDA_DES_PerformSearch931<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 932: CUDA_DES_PerformSearch932<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 933: CUDA_DES_PerformSearch933<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 934: CUDA_DES_PerformSearch934<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 935: CUDA_DES_PerformSearch935<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 936: CUDA_DES_PerformSearch936<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 937: CUDA_DES_PerformSearch937<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 938: CUDA_DES_PerformSearch938<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 939: CUDA_DES_PerformSearch939<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 940: CUDA_DES_PerformSearch940<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 941: CUDA_DES_PerformSearch941<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 942: CUDA_DES_PerformSearch942<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 943: CUDA_DES_PerformSearch943<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 944: CUDA_DES_PerformSearch944<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 945: CUDA_DES_PerformSearch945<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 946: CUDA_DES_PerformSearch946<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 947: CUDA_DES_PerformSearch947<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 948: CUDA_DES_PerformSearch948<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 949: CUDA_DES_PerformSearch949<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 950: CUDA_DES_PerformSearch950<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 951: CUDA_DES_PerformSearch951<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 952: CUDA_DES_PerformSearch952<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 953: CUDA_DES_PerformSearch953<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 954: CUDA_DES_PerformSearch954<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 955: CUDA_DES_PerformSearch955<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 956: CUDA_DES_PerformSearch956<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 957: CUDA_DES_PerformSearch957<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 958: CUDA_DES_PerformSearch958<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 959: CUDA_DES_PerformSearch959<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 960: CUDA_DES_PerformSearch960<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 961: CUDA_DES_PerformSearch961<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 962: CUDA_DES_PerformSearch962<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 963: CUDA_DES_PerformSearch963<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 964: CUDA_DES_PerformSearch964<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 965: CUDA_DES_PerformSearch965<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 966: CUDA_DES_PerformSearch966<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 967: CUDA_DES_PerformSearch967<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 968: CUDA_DES_PerformSearch968<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 969: CUDA_DES_PerformSearch969<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 970: CUDA_DES_PerformSearch970<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 971: CUDA_DES_PerformSearch971<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 972: CUDA_DES_PerformSearch972<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 973: CUDA_DES_PerformSearch973<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 974: CUDA_DES_PerformSearch974<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 975: CUDA_DES_PerformSearch975<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 976: CUDA_DES_PerformSearch976<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 977: CUDA_DES_PerformSearch977<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 978: CUDA_DES_PerformSearch978<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 979: CUDA_DES_PerformSearch979<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 980: CUDA_DES_PerformSearch980<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 981: CUDA_DES_PerformSearch981<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 982: CUDA_DES_PerformSearch982<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 983: CUDA_DES_PerformSearch983<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 984: CUDA_DES_PerformSearch984<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 985: CUDA_DES_PerformSearch985<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 986: CUDA_DES_PerformSearch986<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 987: CUDA_DES_PerformSearch987<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 988: CUDA_DES_PerformSearch988<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 989: CUDA_DES_PerformSearch989<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 990: CUDA_DES_PerformSearch990<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 991: CUDA_DES_PerformSearch991<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 992: CUDA_DES_PerformSearch992<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 993: CUDA_DES_PerformSearch993<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 994: CUDA_DES_PerformSearch994<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 995: CUDA_DES_PerformSearch995<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 996: CUDA_DES_PerformSearch996<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 997: CUDA_DES_PerformSearch997<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 998: CUDA_DES_PerformSearch998<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 999: CUDA_DES_PerformSearch999<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1000: CUDA_DES_PerformSearch1000<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1001: CUDA_DES_PerformSearch1001<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1002: CUDA_DES_PerformSearch1002<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1003: CUDA_DES_PerformSearch1003<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1004: CUDA_DES_PerformSearch1004<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1005: CUDA_DES_PerformSearch1005<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1006: CUDA_DES_PerformSearch1006<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1007: CUDA_DES_PerformSearch1007<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1008: CUDA_DES_PerformSearch1008<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1009: CUDA_DES_PerformSearch1009<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1010: CUDA_DES_PerformSearch1010<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1011: CUDA_DES_PerformSearch1011<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1012: CUDA_DES_PerformSearch1012<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1013: CUDA_DES_PerformSearch1013<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1014: CUDA_DES_PerformSearch1014<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1015: CUDA_DES_PerformSearch1015<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1016: CUDA_DES_PerformSearch1016<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1017: CUDA_DES_PerformSearch1017<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1018: CUDA_DES_PerformSearch1018<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1019: CUDA_DES_PerformSearch1019<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1020: CUDA_DES_PerformSearch1020<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1021: CUDA_DES_PerformSearch1021<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1022: CUDA_DES_PerformSearch1022<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 1023: CUDA_DES_PerformSearch1023<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
