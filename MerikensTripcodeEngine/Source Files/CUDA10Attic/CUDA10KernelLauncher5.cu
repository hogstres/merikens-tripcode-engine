// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 2560
#include "CUDA10Kernel.h"
#define SALT 2561
#include "CUDA10Kernel.h"
#define SALT 2562
#include "CUDA10Kernel.h"
#define SALT 2563
#include "CUDA10Kernel.h"
#define SALT 2564
#include "CUDA10Kernel.h"
#define SALT 2565
#include "CUDA10Kernel.h"
#define SALT 2566
#include "CUDA10Kernel.h"
#define SALT 2567
#include "CUDA10Kernel.h"
#define SALT 2568
#include "CUDA10Kernel.h"
#define SALT 2569
#include "CUDA10Kernel.h"
#define SALT 2570
#include "CUDA10Kernel.h"
#define SALT 2571
#include "CUDA10Kernel.h"
#define SALT 2572
#include "CUDA10Kernel.h"
#define SALT 2573
#include "CUDA10Kernel.h"
#define SALT 2574
#include "CUDA10Kernel.h"
#define SALT 2575
#include "CUDA10Kernel.h"
#define SALT 2576
#include "CUDA10Kernel.h"
#define SALT 2577
#include "CUDA10Kernel.h"
#define SALT 2578
#include "CUDA10Kernel.h"
#define SALT 2579
#include "CUDA10Kernel.h"
#define SALT 2580
#include "CUDA10Kernel.h"
#define SALT 2581
#include "CUDA10Kernel.h"
#define SALT 2582
#include "CUDA10Kernel.h"
#define SALT 2583
#include "CUDA10Kernel.h"
#define SALT 2584
#include "CUDA10Kernel.h"
#define SALT 2585
#include "CUDA10Kernel.h"
#define SALT 2586
#include "CUDA10Kernel.h"
#define SALT 2587
#include "CUDA10Kernel.h"
#define SALT 2588
#include "CUDA10Kernel.h"
#define SALT 2589
#include "CUDA10Kernel.h"
#define SALT 2590
#include "CUDA10Kernel.h"
#define SALT 2591
#include "CUDA10Kernel.h"
#define SALT 2592
#include "CUDA10Kernel.h"
#define SALT 2593
#include "CUDA10Kernel.h"
#define SALT 2594
#include "CUDA10Kernel.h"
#define SALT 2595
#include "CUDA10Kernel.h"
#define SALT 2596
#include "CUDA10Kernel.h"
#define SALT 2597
#include "CUDA10Kernel.h"
#define SALT 2598
#include "CUDA10Kernel.h"
#define SALT 2599
#include "CUDA10Kernel.h"
#define SALT 2600
#include "CUDA10Kernel.h"
#define SALT 2601
#include "CUDA10Kernel.h"
#define SALT 2602
#include "CUDA10Kernel.h"
#define SALT 2603
#include "CUDA10Kernel.h"
#define SALT 2604
#include "CUDA10Kernel.h"
#define SALT 2605
#include "CUDA10Kernel.h"
#define SALT 2606
#include "CUDA10Kernel.h"
#define SALT 2607
#include "CUDA10Kernel.h"
#define SALT 2608
#include "CUDA10Kernel.h"
#define SALT 2609
#include "CUDA10Kernel.h"
#define SALT 2610
#include "CUDA10Kernel.h"
#define SALT 2611
#include "CUDA10Kernel.h"
#define SALT 2612
#include "CUDA10Kernel.h"
#define SALT 2613
#include "CUDA10Kernel.h"
#define SALT 2614
#include "CUDA10Kernel.h"
#define SALT 2615
#include "CUDA10Kernel.h"
#define SALT 2616
#include "CUDA10Kernel.h"
#define SALT 2617
#include "CUDA10Kernel.h"
#define SALT 2618
#include "CUDA10Kernel.h"
#define SALT 2619
#include "CUDA10Kernel.h"
#define SALT 2620
#include "CUDA10Kernel.h"
#define SALT 2621
#include "CUDA10Kernel.h"
#define SALT 2622
#include "CUDA10Kernel.h"
#define SALT 2623
#include "CUDA10Kernel.h"
#define SALT 2624
#include "CUDA10Kernel.h"
#define SALT 2625
#include "CUDA10Kernel.h"
#define SALT 2626
#include "CUDA10Kernel.h"
#define SALT 2627
#include "CUDA10Kernel.h"
#define SALT 2628
#include "CUDA10Kernel.h"
#define SALT 2629
#include "CUDA10Kernel.h"
#define SALT 2630
#include "CUDA10Kernel.h"
#define SALT 2631
#include "CUDA10Kernel.h"
#define SALT 2632
#include "CUDA10Kernel.h"
#define SALT 2633
#include "CUDA10Kernel.h"
#define SALT 2634
#include "CUDA10Kernel.h"
#define SALT 2635
#include "CUDA10Kernel.h"
#define SALT 2636
#include "CUDA10Kernel.h"
#define SALT 2637
#include "CUDA10Kernel.h"
#define SALT 2638
#include "CUDA10Kernel.h"
#define SALT 2639
#include "CUDA10Kernel.h"
#define SALT 2640
#include "CUDA10Kernel.h"
#define SALT 2641
#include "CUDA10Kernel.h"
#define SALT 2642
#include "CUDA10Kernel.h"
#define SALT 2643
#include "CUDA10Kernel.h"
#define SALT 2644
#include "CUDA10Kernel.h"
#define SALT 2645
#include "CUDA10Kernel.h"
#define SALT 2646
#include "CUDA10Kernel.h"
#define SALT 2647
#include "CUDA10Kernel.h"
#define SALT 2648
#include "CUDA10Kernel.h"
#define SALT 2649
#include "CUDA10Kernel.h"
#define SALT 2650
#include "CUDA10Kernel.h"
#define SALT 2651
#include "CUDA10Kernel.h"
#define SALT 2652
#include "CUDA10Kernel.h"
#define SALT 2653
#include "CUDA10Kernel.h"
#define SALT 2654
#include "CUDA10Kernel.h"
#define SALT 2655
#include "CUDA10Kernel.h"
#define SALT 2656
#include "CUDA10Kernel.h"
#define SALT 2657
#include "CUDA10Kernel.h"
#define SALT 2658
#include "CUDA10Kernel.h"
#define SALT 2659
#include "CUDA10Kernel.h"
#define SALT 2660
#include "CUDA10Kernel.h"
#define SALT 2661
#include "CUDA10Kernel.h"
#define SALT 2662
#include "CUDA10Kernel.h"
#define SALT 2663
#include "CUDA10Kernel.h"
#define SALT 2664
#include "CUDA10Kernel.h"
#define SALT 2665
#include "CUDA10Kernel.h"
#define SALT 2666
#include "CUDA10Kernel.h"
#define SALT 2667
#include "CUDA10Kernel.h"
#define SALT 2668
#include "CUDA10Kernel.h"
#define SALT 2669
#include "CUDA10Kernel.h"
#define SALT 2670
#include "CUDA10Kernel.h"
#define SALT 2671
#include "CUDA10Kernel.h"
#define SALT 2672
#include "CUDA10Kernel.h"
#define SALT 2673
#include "CUDA10Kernel.h"
#define SALT 2674
#include "CUDA10Kernel.h"
#define SALT 2675
#include "CUDA10Kernel.h"
#define SALT 2676
#include "CUDA10Kernel.h"
#define SALT 2677
#include "CUDA10Kernel.h"
#define SALT 2678
#include "CUDA10Kernel.h"
#define SALT 2679
#include "CUDA10Kernel.h"
#define SALT 2680
#include "CUDA10Kernel.h"
#define SALT 2681
#include "CUDA10Kernel.h"
#define SALT 2682
#include "CUDA10Kernel.h"
#define SALT 2683
#include "CUDA10Kernel.h"
#define SALT 2684
#include "CUDA10Kernel.h"
#define SALT 2685
#include "CUDA10Kernel.h"
#define SALT 2686
#include "CUDA10Kernel.h"
#define SALT 2687
#include "CUDA10Kernel.h"
#define SALT 2688
#include "CUDA10Kernel.h"
#define SALT 2689
#include "CUDA10Kernel.h"
#define SALT 2690
#include "CUDA10Kernel.h"
#define SALT 2691
#include "CUDA10Kernel.h"
#define SALT 2692
#include "CUDA10Kernel.h"
#define SALT 2693
#include "CUDA10Kernel.h"
#define SALT 2694
#include "CUDA10Kernel.h"
#define SALT 2695
#include "CUDA10Kernel.h"
#define SALT 2696
#include "CUDA10Kernel.h"
#define SALT 2697
#include "CUDA10Kernel.h"
#define SALT 2698
#include "CUDA10Kernel.h"
#define SALT 2699
#include "CUDA10Kernel.h"
#define SALT 2700
#include "CUDA10Kernel.h"
#define SALT 2701
#include "CUDA10Kernel.h"
#define SALT 2702
#include "CUDA10Kernel.h"
#define SALT 2703
#include "CUDA10Kernel.h"
#define SALT 2704
#include "CUDA10Kernel.h"
#define SALT 2705
#include "CUDA10Kernel.h"
#define SALT 2706
#include "CUDA10Kernel.h"
#define SALT 2707
#include "CUDA10Kernel.h"
#define SALT 2708
#include "CUDA10Kernel.h"
#define SALT 2709
#include "CUDA10Kernel.h"
#define SALT 2710
#include "CUDA10Kernel.h"
#define SALT 2711
#include "CUDA10Kernel.h"
#define SALT 2712
#include "CUDA10Kernel.h"
#define SALT 2713
#include "CUDA10Kernel.h"
#define SALT 2714
#include "CUDA10Kernel.h"
#define SALT 2715
#include "CUDA10Kernel.h"
#define SALT 2716
#include "CUDA10Kernel.h"
#define SALT 2717
#include "CUDA10Kernel.h"
#define SALT 2718
#include "CUDA10Kernel.h"
#define SALT 2719
#include "CUDA10Kernel.h"
#define SALT 2720
#include "CUDA10Kernel.h"
#define SALT 2721
#include "CUDA10Kernel.h"
#define SALT 2722
#include "CUDA10Kernel.h"
#define SALT 2723
#include "CUDA10Kernel.h"
#define SALT 2724
#include "CUDA10Kernel.h"
#define SALT 2725
#include "CUDA10Kernel.h"
#define SALT 2726
#include "CUDA10Kernel.h"
#define SALT 2727
#include "CUDA10Kernel.h"
#define SALT 2728
#include "CUDA10Kernel.h"
#define SALT 2729
#include "CUDA10Kernel.h"
#define SALT 2730
#include "CUDA10Kernel.h"
#define SALT 2731
#include "CUDA10Kernel.h"
#define SALT 2732
#include "CUDA10Kernel.h"
#define SALT 2733
#include "CUDA10Kernel.h"
#define SALT 2734
#include "CUDA10Kernel.h"
#define SALT 2735
#include "CUDA10Kernel.h"
#define SALT 2736
#include "CUDA10Kernel.h"
#define SALT 2737
#include "CUDA10Kernel.h"
#define SALT 2738
#include "CUDA10Kernel.h"
#define SALT 2739
#include "CUDA10Kernel.h"
#define SALT 2740
#include "CUDA10Kernel.h"
#define SALT 2741
#include "CUDA10Kernel.h"
#define SALT 2742
#include "CUDA10Kernel.h"
#define SALT 2743
#include "CUDA10Kernel.h"
#define SALT 2744
#include "CUDA10Kernel.h"
#define SALT 2745
#include "CUDA10Kernel.h"
#define SALT 2746
#include "CUDA10Kernel.h"
#define SALT 2747
#include "CUDA10Kernel.h"
#define SALT 2748
#include "CUDA10Kernel.h"
#define SALT 2749
#include "CUDA10Kernel.h"
#define SALT 2750
#include "CUDA10Kernel.h"
#define SALT 2751
#include "CUDA10Kernel.h"
#define SALT 2752
#include "CUDA10Kernel.h"
#define SALT 2753
#include "CUDA10Kernel.h"
#define SALT 2754
#include "CUDA10Kernel.h"
#define SALT 2755
#include "CUDA10Kernel.h"
#define SALT 2756
#include "CUDA10Kernel.h"
#define SALT 2757
#include "CUDA10Kernel.h"
#define SALT 2758
#include "CUDA10Kernel.h"
#define SALT 2759
#include "CUDA10Kernel.h"
#define SALT 2760
#include "CUDA10Kernel.h"
#define SALT 2761
#include "CUDA10Kernel.h"
#define SALT 2762
#include "CUDA10Kernel.h"
#define SALT 2763
#include "CUDA10Kernel.h"
#define SALT 2764
#include "CUDA10Kernel.h"
#define SALT 2765
#include "CUDA10Kernel.h"
#define SALT 2766
#include "CUDA10Kernel.h"
#define SALT 2767
#include "CUDA10Kernel.h"
#define SALT 2768
#include "CUDA10Kernel.h"
#define SALT 2769
#include "CUDA10Kernel.h"
#define SALT 2770
#include "CUDA10Kernel.h"
#define SALT 2771
#include "CUDA10Kernel.h"
#define SALT 2772
#include "CUDA10Kernel.h"
#define SALT 2773
#include "CUDA10Kernel.h"
#define SALT 2774
#include "CUDA10Kernel.h"
#define SALT 2775
#include "CUDA10Kernel.h"
#define SALT 2776
#include "CUDA10Kernel.h"
#define SALT 2777
#include "CUDA10Kernel.h"
#define SALT 2778
#include "CUDA10Kernel.h"
#define SALT 2779
#include "CUDA10Kernel.h"
#define SALT 2780
#include "CUDA10Kernel.h"
#define SALT 2781
#include "CUDA10Kernel.h"
#define SALT 2782
#include "CUDA10Kernel.h"
#define SALT 2783
#include "CUDA10Kernel.h"
#define SALT 2784
#include "CUDA10Kernel.h"
#define SALT 2785
#include "CUDA10Kernel.h"
#define SALT 2786
#include "CUDA10Kernel.h"
#define SALT 2787
#include "CUDA10Kernel.h"
#define SALT 2788
#include "CUDA10Kernel.h"
#define SALT 2789
#include "CUDA10Kernel.h"
#define SALT 2790
#include "CUDA10Kernel.h"
#define SALT 2791
#include "CUDA10Kernel.h"
#define SALT 2792
#include "CUDA10Kernel.h"
#define SALT 2793
#include "CUDA10Kernel.h"
#define SALT 2794
#include "CUDA10Kernel.h"
#define SALT 2795
#include "CUDA10Kernel.h"
#define SALT 2796
#include "CUDA10Kernel.h"
#define SALT 2797
#include "CUDA10Kernel.h"
#define SALT 2798
#include "CUDA10Kernel.h"
#define SALT 2799
#include "CUDA10Kernel.h"
#define SALT 2800
#include "CUDA10Kernel.h"
#define SALT 2801
#include "CUDA10Kernel.h"
#define SALT 2802
#include "CUDA10Kernel.h"
#define SALT 2803
#include "CUDA10Kernel.h"
#define SALT 2804
#include "CUDA10Kernel.h"
#define SALT 2805
#include "CUDA10Kernel.h"
#define SALT 2806
#include "CUDA10Kernel.h"
#define SALT 2807
#include "CUDA10Kernel.h"
#define SALT 2808
#include "CUDA10Kernel.h"
#define SALT 2809
#include "CUDA10Kernel.h"
#define SALT 2810
#include "CUDA10Kernel.h"
#define SALT 2811
#include "CUDA10Kernel.h"
#define SALT 2812
#include "CUDA10Kernel.h"
#define SALT 2813
#include "CUDA10Kernel.h"
#define SALT 2814
#include "CUDA10Kernel.h"
#define SALT 2815
#include "CUDA10Kernel.h"
#define SALT 2816
#include "CUDA10Kernel.h"
#define SALT 2817
#include "CUDA10Kernel.h"
#define SALT 2818
#include "CUDA10Kernel.h"
#define SALT 2819
#include "CUDA10Kernel.h"
#define SALT 2820
#include "CUDA10Kernel.h"
#define SALT 2821
#include "CUDA10Kernel.h"
#define SALT 2822
#include "CUDA10Kernel.h"
#define SALT 2823
#include "CUDA10Kernel.h"
#define SALT 2824
#include "CUDA10Kernel.h"
#define SALT 2825
#include "CUDA10Kernel.h"
#define SALT 2826
#include "CUDA10Kernel.h"
#define SALT 2827
#include "CUDA10Kernel.h"
#define SALT 2828
#include "CUDA10Kernel.h"
#define SALT 2829
#include "CUDA10Kernel.h"
#define SALT 2830
#include "CUDA10Kernel.h"
#define SALT 2831
#include "CUDA10Kernel.h"
#define SALT 2832
#include "CUDA10Kernel.h"
#define SALT 2833
#include "CUDA10Kernel.h"
#define SALT 2834
#include "CUDA10Kernel.h"
#define SALT 2835
#include "CUDA10Kernel.h"
#define SALT 2836
#include "CUDA10Kernel.h"
#define SALT 2837
#include "CUDA10Kernel.h"
#define SALT 2838
#include "CUDA10Kernel.h"
#define SALT 2839
#include "CUDA10Kernel.h"
#define SALT 2840
#include "CUDA10Kernel.h"
#define SALT 2841
#include "CUDA10Kernel.h"
#define SALT 2842
#include "CUDA10Kernel.h"
#define SALT 2843
#include "CUDA10Kernel.h"
#define SALT 2844
#include "CUDA10Kernel.h"
#define SALT 2845
#include "CUDA10Kernel.h"
#define SALT 2846
#include "CUDA10Kernel.h"
#define SALT 2847
#include "CUDA10Kernel.h"
#define SALT 2848
#include "CUDA10Kernel.h"
#define SALT 2849
#include "CUDA10Kernel.h"
#define SALT 2850
#include "CUDA10Kernel.h"
#define SALT 2851
#include "CUDA10Kernel.h"
#define SALT 2852
#include "CUDA10Kernel.h"
#define SALT 2853
#include "CUDA10Kernel.h"
#define SALT 2854
#include "CUDA10Kernel.h"
#define SALT 2855
#include "CUDA10Kernel.h"
#define SALT 2856
#include "CUDA10Kernel.h"
#define SALT 2857
#include "CUDA10Kernel.h"
#define SALT 2858
#include "CUDA10Kernel.h"
#define SALT 2859
#include "CUDA10Kernel.h"
#define SALT 2860
#include "CUDA10Kernel.h"
#define SALT 2861
#include "CUDA10Kernel.h"
#define SALT 2862
#include "CUDA10Kernel.h"
#define SALT 2863
#include "CUDA10Kernel.h"
#define SALT 2864
#include "CUDA10Kernel.h"
#define SALT 2865
#include "CUDA10Kernel.h"
#define SALT 2866
#include "CUDA10Kernel.h"
#define SALT 2867
#include "CUDA10Kernel.h"
#define SALT 2868
#include "CUDA10Kernel.h"
#define SALT 2869
#include "CUDA10Kernel.h"
#define SALT 2870
#include "CUDA10Kernel.h"
#define SALT 2871
#include "CUDA10Kernel.h"
#define SALT 2872
#include "CUDA10Kernel.h"
#define SALT 2873
#include "CUDA10Kernel.h"
#define SALT 2874
#include "CUDA10Kernel.h"
#define SALT 2875
#include "CUDA10Kernel.h"
#define SALT 2876
#include "CUDA10Kernel.h"
#define SALT 2877
#include "CUDA10Kernel.h"
#define SALT 2878
#include "CUDA10Kernel.h"
#define SALT 2879
#include "CUDA10Kernel.h"
#define SALT 2880
#include "CUDA10Kernel.h"
#define SALT 2881
#include "CUDA10Kernel.h"
#define SALT 2882
#include "CUDA10Kernel.h"
#define SALT 2883
#include "CUDA10Kernel.h"
#define SALT 2884
#include "CUDA10Kernel.h"
#define SALT 2885
#include "CUDA10Kernel.h"
#define SALT 2886
#include "CUDA10Kernel.h"
#define SALT 2887
#include "CUDA10Kernel.h"
#define SALT 2888
#include "CUDA10Kernel.h"
#define SALT 2889
#include "CUDA10Kernel.h"
#define SALT 2890
#include "CUDA10Kernel.h"
#define SALT 2891
#include "CUDA10Kernel.h"
#define SALT 2892
#include "CUDA10Kernel.h"
#define SALT 2893
#include "CUDA10Kernel.h"
#define SALT 2894
#include "CUDA10Kernel.h"
#define SALT 2895
#include "CUDA10Kernel.h"
#define SALT 2896
#include "CUDA10Kernel.h"
#define SALT 2897
#include "CUDA10Kernel.h"
#define SALT 2898
#include "CUDA10Kernel.h"
#define SALT 2899
#include "CUDA10Kernel.h"
#define SALT 2900
#include "CUDA10Kernel.h"
#define SALT 2901
#include "CUDA10Kernel.h"
#define SALT 2902
#include "CUDA10Kernel.h"
#define SALT 2903
#include "CUDA10Kernel.h"
#define SALT 2904
#include "CUDA10Kernel.h"
#define SALT 2905
#include "CUDA10Kernel.h"
#define SALT 2906
#include "CUDA10Kernel.h"
#define SALT 2907
#include "CUDA10Kernel.h"
#define SALT 2908
#include "CUDA10Kernel.h"
#define SALT 2909
#include "CUDA10Kernel.h"
#define SALT 2910
#include "CUDA10Kernel.h"
#define SALT 2911
#include "CUDA10Kernel.h"
#define SALT 2912
#include "CUDA10Kernel.h"
#define SALT 2913
#include "CUDA10Kernel.h"
#define SALT 2914
#include "CUDA10Kernel.h"
#define SALT 2915
#include "CUDA10Kernel.h"
#define SALT 2916
#include "CUDA10Kernel.h"
#define SALT 2917
#include "CUDA10Kernel.h"
#define SALT 2918
#include "CUDA10Kernel.h"
#define SALT 2919
#include "CUDA10Kernel.h"
#define SALT 2920
#include "CUDA10Kernel.h"
#define SALT 2921
#include "CUDA10Kernel.h"
#define SALT 2922
#include "CUDA10Kernel.h"
#define SALT 2923
#include "CUDA10Kernel.h"
#define SALT 2924
#include "CUDA10Kernel.h"
#define SALT 2925
#include "CUDA10Kernel.h"
#define SALT 2926
#include "CUDA10Kernel.h"
#define SALT 2927
#include "CUDA10Kernel.h"
#define SALT 2928
#include "CUDA10Kernel.h"
#define SALT 2929
#include "CUDA10Kernel.h"
#define SALT 2930
#include "CUDA10Kernel.h"
#define SALT 2931
#include "CUDA10Kernel.h"
#define SALT 2932
#include "CUDA10Kernel.h"
#define SALT 2933
#include "CUDA10Kernel.h"
#define SALT 2934
#include "CUDA10Kernel.h"
#define SALT 2935
#include "CUDA10Kernel.h"
#define SALT 2936
#include "CUDA10Kernel.h"
#define SALT 2937
#include "CUDA10Kernel.h"
#define SALT 2938
#include "CUDA10Kernel.h"
#define SALT 2939
#include "CUDA10Kernel.h"
#define SALT 2940
#include "CUDA10Kernel.h"
#define SALT 2941
#include "CUDA10Kernel.h"
#define SALT 2942
#include "CUDA10Kernel.h"
#define SALT 2943
#include "CUDA10Kernel.h"
#define SALT 2944
#include "CUDA10Kernel.h"
#define SALT 2945
#include "CUDA10Kernel.h"
#define SALT 2946
#include "CUDA10Kernel.h"
#define SALT 2947
#include "CUDA10Kernel.h"
#define SALT 2948
#include "CUDA10Kernel.h"
#define SALT 2949
#include "CUDA10Kernel.h"
#define SALT 2950
#include "CUDA10Kernel.h"
#define SALT 2951
#include "CUDA10Kernel.h"
#define SALT 2952
#include "CUDA10Kernel.h"
#define SALT 2953
#include "CUDA10Kernel.h"
#define SALT 2954
#include "CUDA10Kernel.h"
#define SALT 2955
#include "CUDA10Kernel.h"
#define SALT 2956
#include "CUDA10Kernel.h"
#define SALT 2957
#include "CUDA10Kernel.h"
#define SALT 2958
#include "CUDA10Kernel.h"
#define SALT 2959
#include "CUDA10Kernel.h"
#define SALT 2960
#include "CUDA10Kernel.h"
#define SALT 2961
#include "CUDA10Kernel.h"
#define SALT 2962
#include "CUDA10Kernel.h"
#define SALT 2963
#include "CUDA10Kernel.h"
#define SALT 2964
#include "CUDA10Kernel.h"
#define SALT 2965
#include "CUDA10Kernel.h"
#define SALT 2966
#include "CUDA10Kernel.h"
#define SALT 2967
#include "CUDA10Kernel.h"
#define SALT 2968
#include "CUDA10Kernel.h"
#define SALT 2969
#include "CUDA10Kernel.h"
#define SALT 2970
#include "CUDA10Kernel.h"
#define SALT 2971
#include "CUDA10Kernel.h"
#define SALT 2972
#include "CUDA10Kernel.h"
#define SALT 2973
#include "CUDA10Kernel.h"
#define SALT 2974
#include "CUDA10Kernel.h"
#define SALT 2975
#include "CUDA10Kernel.h"
#define SALT 2976
#include "CUDA10Kernel.h"
#define SALT 2977
#include "CUDA10Kernel.h"
#define SALT 2978
#include "CUDA10Kernel.h"
#define SALT 2979
#include "CUDA10Kernel.h"
#define SALT 2980
#include "CUDA10Kernel.h"
#define SALT 2981
#include "CUDA10Kernel.h"
#define SALT 2982
#include "CUDA10Kernel.h"
#define SALT 2983
#include "CUDA10Kernel.h"
#define SALT 2984
#include "CUDA10Kernel.h"
#define SALT 2985
#include "CUDA10Kernel.h"
#define SALT 2986
#include "CUDA10Kernel.h"
#define SALT 2987
#include "CUDA10Kernel.h"
#define SALT 2988
#include "CUDA10Kernel.h"
#define SALT 2989
#include "CUDA10Kernel.h"
#define SALT 2990
#include "CUDA10Kernel.h"
#define SALT 2991
#include "CUDA10Kernel.h"
#define SALT 2992
#include "CUDA10Kernel.h"
#define SALT 2993
#include "CUDA10Kernel.h"
#define SALT 2994
#include "CUDA10Kernel.h"
#define SALT 2995
#include "CUDA10Kernel.h"
#define SALT 2996
#include "CUDA10Kernel.h"
#define SALT 2997
#include "CUDA10Kernel.h"
#define SALT 2998
#include "CUDA10Kernel.h"
#define SALT 2999
#include "CUDA10Kernel.h"
#define SALT 3000
#include "CUDA10Kernel.h"
#define SALT 3001
#include "CUDA10Kernel.h"
#define SALT 3002
#include "CUDA10Kernel.h"
#define SALT 3003
#include "CUDA10Kernel.h"
#define SALT 3004
#include "CUDA10Kernel.h"
#define SALT 3005
#include "CUDA10Kernel.h"
#define SALT 3006
#include "CUDA10Kernel.h"
#define SALT 3007
#include "CUDA10Kernel.h"
#define SALT 3008
#include "CUDA10Kernel.h"
#define SALT 3009
#include "CUDA10Kernel.h"
#define SALT 3010
#include "CUDA10Kernel.h"
#define SALT 3011
#include "CUDA10Kernel.h"
#define SALT 3012
#include "CUDA10Kernel.h"
#define SALT 3013
#include "CUDA10Kernel.h"
#define SALT 3014
#include "CUDA10Kernel.h"
#define SALT 3015
#include "CUDA10Kernel.h"
#define SALT 3016
#include "CUDA10Kernel.h"
#define SALT 3017
#include "CUDA10Kernel.h"
#define SALT 3018
#include "CUDA10Kernel.h"
#define SALT 3019
#include "CUDA10Kernel.h"
#define SALT 3020
#include "CUDA10Kernel.h"
#define SALT 3021
#include "CUDA10Kernel.h"
#define SALT 3022
#include "CUDA10Kernel.h"
#define SALT 3023
#include "CUDA10Kernel.h"
#define SALT 3024
#include "CUDA10Kernel.h"
#define SALT 3025
#include "CUDA10Kernel.h"
#define SALT 3026
#include "CUDA10Kernel.h"
#define SALT 3027
#include "CUDA10Kernel.h"
#define SALT 3028
#include "CUDA10Kernel.h"
#define SALT 3029
#include "CUDA10Kernel.h"
#define SALT 3030
#include "CUDA10Kernel.h"
#define SALT 3031
#include "CUDA10Kernel.h"
#define SALT 3032
#include "CUDA10Kernel.h"
#define SALT 3033
#include "CUDA10Kernel.h"
#define SALT 3034
#include "CUDA10Kernel.h"
#define SALT 3035
#include "CUDA10Kernel.h"
#define SALT 3036
#include "CUDA10Kernel.h"
#define SALT 3037
#include "CUDA10Kernel.h"
#define SALT 3038
#include "CUDA10Kernel.h"
#define SALT 3039
#include "CUDA10Kernel.h"
#define SALT 3040
#include "CUDA10Kernel.h"
#define SALT 3041
#include "CUDA10Kernel.h"
#define SALT 3042
#include "CUDA10Kernel.h"
#define SALT 3043
#include "CUDA10Kernel.h"
#define SALT 3044
#include "CUDA10Kernel.h"
#define SALT 3045
#include "CUDA10Kernel.h"
#define SALT 3046
#include "CUDA10Kernel.h"
#define SALT 3047
#include "CUDA10Kernel.h"
#define SALT 3048
#include "CUDA10Kernel.h"
#define SALT 3049
#include "CUDA10Kernel.h"
#define SALT 3050
#include "CUDA10Kernel.h"
#define SALT 3051
#include "CUDA10Kernel.h"
#define SALT 3052
#include "CUDA10Kernel.h"
#define SALT 3053
#include "CUDA10Kernel.h"
#define SALT 3054
#include "CUDA10Kernel.h"
#define SALT 3055
#include "CUDA10Kernel.h"
#define SALT 3056
#include "CUDA10Kernel.h"
#define SALT 3057
#include "CUDA10Kernel.h"
#define SALT 3058
#include "CUDA10Kernel.h"
#define SALT 3059
#include "CUDA10Kernel.h"
#define SALT 3060
#include "CUDA10Kernel.h"
#define SALT 3061
#include "CUDA10Kernel.h"
#define SALT 3062
#include "CUDA10Kernel.h"
#define SALT 3063
#include "CUDA10Kernel.h"
#define SALT 3064
#include "CUDA10Kernel.h"
#define SALT 3065
#include "CUDA10Kernel.h"
#define SALT 3066
#include "CUDA10Kernel.h"
#define SALT 3067
#include "CUDA10Kernel.h"
#define SALT 3068
#include "CUDA10Kernel.h"
#define SALT 3069
#include "CUDA10Kernel.h"
#define SALT 3070
#include "CUDA10Kernel.h"
#define SALT 3071
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher5()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel5(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 2560: CUDA_DES_PerformSearch2560<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2561: CUDA_DES_PerformSearch2561<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2562: CUDA_DES_PerformSearch2562<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2563: CUDA_DES_PerformSearch2563<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2564: CUDA_DES_PerformSearch2564<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2565: CUDA_DES_PerformSearch2565<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2566: CUDA_DES_PerformSearch2566<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2567: CUDA_DES_PerformSearch2567<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2568: CUDA_DES_PerformSearch2568<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2569: CUDA_DES_PerformSearch2569<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2570: CUDA_DES_PerformSearch2570<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2571: CUDA_DES_PerformSearch2571<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2572: CUDA_DES_PerformSearch2572<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2573: CUDA_DES_PerformSearch2573<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2574: CUDA_DES_PerformSearch2574<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2575: CUDA_DES_PerformSearch2575<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2576: CUDA_DES_PerformSearch2576<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2577: CUDA_DES_PerformSearch2577<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2578: CUDA_DES_PerformSearch2578<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2579: CUDA_DES_PerformSearch2579<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2580: CUDA_DES_PerformSearch2580<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2581: CUDA_DES_PerformSearch2581<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2582: CUDA_DES_PerformSearch2582<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2583: CUDA_DES_PerformSearch2583<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2584: CUDA_DES_PerformSearch2584<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2585: CUDA_DES_PerformSearch2585<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2586: CUDA_DES_PerformSearch2586<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2587: CUDA_DES_PerformSearch2587<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2588: CUDA_DES_PerformSearch2588<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2589: CUDA_DES_PerformSearch2589<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2590: CUDA_DES_PerformSearch2590<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2591: CUDA_DES_PerformSearch2591<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2592: CUDA_DES_PerformSearch2592<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2593: CUDA_DES_PerformSearch2593<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2594: CUDA_DES_PerformSearch2594<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2595: CUDA_DES_PerformSearch2595<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2596: CUDA_DES_PerformSearch2596<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2597: CUDA_DES_PerformSearch2597<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2598: CUDA_DES_PerformSearch2598<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2599: CUDA_DES_PerformSearch2599<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2600: CUDA_DES_PerformSearch2600<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2601: CUDA_DES_PerformSearch2601<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2602: CUDA_DES_PerformSearch2602<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2603: CUDA_DES_PerformSearch2603<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2604: CUDA_DES_PerformSearch2604<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2605: CUDA_DES_PerformSearch2605<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2606: CUDA_DES_PerformSearch2606<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2607: CUDA_DES_PerformSearch2607<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2608: CUDA_DES_PerformSearch2608<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2609: CUDA_DES_PerformSearch2609<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2610: CUDA_DES_PerformSearch2610<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2611: CUDA_DES_PerformSearch2611<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2612: CUDA_DES_PerformSearch2612<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2613: CUDA_DES_PerformSearch2613<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2614: CUDA_DES_PerformSearch2614<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2615: CUDA_DES_PerformSearch2615<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2616: CUDA_DES_PerformSearch2616<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2617: CUDA_DES_PerformSearch2617<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2618: CUDA_DES_PerformSearch2618<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2619: CUDA_DES_PerformSearch2619<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2620: CUDA_DES_PerformSearch2620<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2621: CUDA_DES_PerformSearch2621<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2622: CUDA_DES_PerformSearch2622<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2623: CUDA_DES_PerformSearch2623<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2624: CUDA_DES_PerformSearch2624<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2625: CUDA_DES_PerformSearch2625<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2626: CUDA_DES_PerformSearch2626<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2627: CUDA_DES_PerformSearch2627<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2628: CUDA_DES_PerformSearch2628<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2629: CUDA_DES_PerformSearch2629<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2630: CUDA_DES_PerformSearch2630<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2631: CUDA_DES_PerformSearch2631<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2632: CUDA_DES_PerformSearch2632<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2633: CUDA_DES_PerformSearch2633<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2634: CUDA_DES_PerformSearch2634<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2635: CUDA_DES_PerformSearch2635<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2636: CUDA_DES_PerformSearch2636<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2637: CUDA_DES_PerformSearch2637<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2638: CUDA_DES_PerformSearch2638<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2639: CUDA_DES_PerformSearch2639<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2640: CUDA_DES_PerformSearch2640<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2641: CUDA_DES_PerformSearch2641<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2642: CUDA_DES_PerformSearch2642<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2643: CUDA_DES_PerformSearch2643<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2644: CUDA_DES_PerformSearch2644<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2645: CUDA_DES_PerformSearch2645<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2646: CUDA_DES_PerformSearch2646<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2647: CUDA_DES_PerformSearch2647<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2648: CUDA_DES_PerformSearch2648<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2649: CUDA_DES_PerformSearch2649<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2650: CUDA_DES_PerformSearch2650<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2651: CUDA_DES_PerformSearch2651<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2652: CUDA_DES_PerformSearch2652<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2653: CUDA_DES_PerformSearch2653<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2654: CUDA_DES_PerformSearch2654<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2655: CUDA_DES_PerformSearch2655<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2656: CUDA_DES_PerformSearch2656<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2657: CUDA_DES_PerformSearch2657<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2658: CUDA_DES_PerformSearch2658<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2659: CUDA_DES_PerformSearch2659<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2660: CUDA_DES_PerformSearch2660<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2661: CUDA_DES_PerformSearch2661<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2662: CUDA_DES_PerformSearch2662<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2663: CUDA_DES_PerformSearch2663<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2664: CUDA_DES_PerformSearch2664<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2665: CUDA_DES_PerformSearch2665<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2666: CUDA_DES_PerformSearch2666<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2667: CUDA_DES_PerformSearch2667<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2668: CUDA_DES_PerformSearch2668<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2669: CUDA_DES_PerformSearch2669<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2670: CUDA_DES_PerformSearch2670<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2671: CUDA_DES_PerformSearch2671<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2672: CUDA_DES_PerformSearch2672<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2673: CUDA_DES_PerformSearch2673<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2674: CUDA_DES_PerformSearch2674<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2675: CUDA_DES_PerformSearch2675<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2676: CUDA_DES_PerformSearch2676<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2677: CUDA_DES_PerformSearch2677<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2678: CUDA_DES_PerformSearch2678<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2679: CUDA_DES_PerformSearch2679<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2680: CUDA_DES_PerformSearch2680<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2681: CUDA_DES_PerformSearch2681<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2682: CUDA_DES_PerformSearch2682<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2683: CUDA_DES_PerformSearch2683<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2684: CUDA_DES_PerformSearch2684<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2685: CUDA_DES_PerformSearch2685<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2686: CUDA_DES_PerformSearch2686<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2687: CUDA_DES_PerformSearch2687<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2688: CUDA_DES_PerformSearch2688<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2689: CUDA_DES_PerformSearch2689<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2690: CUDA_DES_PerformSearch2690<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2691: CUDA_DES_PerformSearch2691<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2692: CUDA_DES_PerformSearch2692<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2693: CUDA_DES_PerformSearch2693<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2694: CUDA_DES_PerformSearch2694<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2695: CUDA_DES_PerformSearch2695<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2696: CUDA_DES_PerformSearch2696<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2697: CUDA_DES_PerformSearch2697<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2698: CUDA_DES_PerformSearch2698<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2699: CUDA_DES_PerformSearch2699<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2700: CUDA_DES_PerformSearch2700<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2701: CUDA_DES_PerformSearch2701<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2702: CUDA_DES_PerformSearch2702<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2703: CUDA_DES_PerformSearch2703<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2704: CUDA_DES_PerformSearch2704<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2705: CUDA_DES_PerformSearch2705<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2706: CUDA_DES_PerformSearch2706<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2707: CUDA_DES_PerformSearch2707<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2708: CUDA_DES_PerformSearch2708<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2709: CUDA_DES_PerformSearch2709<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2710: CUDA_DES_PerformSearch2710<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2711: CUDA_DES_PerformSearch2711<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2712: CUDA_DES_PerformSearch2712<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2713: CUDA_DES_PerformSearch2713<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2714: CUDA_DES_PerformSearch2714<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2715: CUDA_DES_PerformSearch2715<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2716: CUDA_DES_PerformSearch2716<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2717: CUDA_DES_PerformSearch2717<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2718: CUDA_DES_PerformSearch2718<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2719: CUDA_DES_PerformSearch2719<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2720: CUDA_DES_PerformSearch2720<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2721: CUDA_DES_PerformSearch2721<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2722: CUDA_DES_PerformSearch2722<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2723: CUDA_DES_PerformSearch2723<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2724: CUDA_DES_PerformSearch2724<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2725: CUDA_DES_PerformSearch2725<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2726: CUDA_DES_PerformSearch2726<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2727: CUDA_DES_PerformSearch2727<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2728: CUDA_DES_PerformSearch2728<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2729: CUDA_DES_PerformSearch2729<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2730: CUDA_DES_PerformSearch2730<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2731: CUDA_DES_PerformSearch2731<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2732: CUDA_DES_PerformSearch2732<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2733: CUDA_DES_PerformSearch2733<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2734: CUDA_DES_PerformSearch2734<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2735: CUDA_DES_PerformSearch2735<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2736: CUDA_DES_PerformSearch2736<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2737: CUDA_DES_PerformSearch2737<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2738: CUDA_DES_PerformSearch2738<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2739: CUDA_DES_PerformSearch2739<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2740: CUDA_DES_PerformSearch2740<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2741: CUDA_DES_PerformSearch2741<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2742: CUDA_DES_PerformSearch2742<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2743: CUDA_DES_PerformSearch2743<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2744: CUDA_DES_PerformSearch2744<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2745: CUDA_DES_PerformSearch2745<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2746: CUDA_DES_PerformSearch2746<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2747: CUDA_DES_PerformSearch2747<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2748: CUDA_DES_PerformSearch2748<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2749: CUDA_DES_PerformSearch2749<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2750: CUDA_DES_PerformSearch2750<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2751: CUDA_DES_PerformSearch2751<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2752: CUDA_DES_PerformSearch2752<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2753: CUDA_DES_PerformSearch2753<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2754: CUDA_DES_PerformSearch2754<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2755: CUDA_DES_PerformSearch2755<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2756: CUDA_DES_PerformSearch2756<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2757: CUDA_DES_PerformSearch2757<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2758: CUDA_DES_PerformSearch2758<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2759: CUDA_DES_PerformSearch2759<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2760: CUDA_DES_PerformSearch2760<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2761: CUDA_DES_PerformSearch2761<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2762: CUDA_DES_PerformSearch2762<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2763: CUDA_DES_PerformSearch2763<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2764: CUDA_DES_PerformSearch2764<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2765: CUDA_DES_PerformSearch2765<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2766: CUDA_DES_PerformSearch2766<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2767: CUDA_DES_PerformSearch2767<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2768: CUDA_DES_PerformSearch2768<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2769: CUDA_DES_PerformSearch2769<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2770: CUDA_DES_PerformSearch2770<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2771: CUDA_DES_PerformSearch2771<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2772: CUDA_DES_PerformSearch2772<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2773: CUDA_DES_PerformSearch2773<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2774: CUDA_DES_PerformSearch2774<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2775: CUDA_DES_PerformSearch2775<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2776: CUDA_DES_PerformSearch2776<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2777: CUDA_DES_PerformSearch2777<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2778: CUDA_DES_PerformSearch2778<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2779: CUDA_DES_PerformSearch2779<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2780: CUDA_DES_PerformSearch2780<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2781: CUDA_DES_PerformSearch2781<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2782: CUDA_DES_PerformSearch2782<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2783: CUDA_DES_PerformSearch2783<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2784: CUDA_DES_PerformSearch2784<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2785: CUDA_DES_PerformSearch2785<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2786: CUDA_DES_PerformSearch2786<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2787: CUDA_DES_PerformSearch2787<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2788: CUDA_DES_PerformSearch2788<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2789: CUDA_DES_PerformSearch2789<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2790: CUDA_DES_PerformSearch2790<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2791: CUDA_DES_PerformSearch2791<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2792: CUDA_DES_PerformSearch2792<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2793: CUDA_DES_PerformSearch2793<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2794: CUDA_DES_PerformSearch2794<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2795: CUDA_DES_PerformSearch2795<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2796: CUDA_DES_PerformSearch2796<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2797: CUDA_DES_PerformSearch2797<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2798: CUDA_DES_PerformSearch2798<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2799: CUDA_DES_PerformSearch2799<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2800: CUDA_DES_PerformSearch2800<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2801: CUDA_DES_PerformSearch2801<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2802: CUDA_DES_PerformSearch2802<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2803: CUDA_DES_PerformSearch2803<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2804: CUDA_DES_PerformSearch2804<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2805: CUDA_DES_PerformSearch2805<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2806: CUDA_DES_PerformSearch2806<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2807: CUDA_DES_PerformSearch2807<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2808: CUDA_DES_PerformSearch2808<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2809: CUDA_DES_PerformSearch2809<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2810: CUDA_DES_PerformSearch2810<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2811: CUDA_DES_PerformSearch2811<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2812: CUDA_DES_PerformSearch2812<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2813: CUDA_DES_PerformSearch2813<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2814: CUDA_DES_PerformSearch2814<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2815: CUDA_DES_PerformSearch2815<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2816: CUDA_DES_PerformSearch2816<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2817: CUDA_DES_PerformSearch2817<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2818: CUDA_DES_PerformSearch2818<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2819: CUDA_DES_PerformSearch2819<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2820: CUDA_DES_PerformSearch2820<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2821: CUDA_DES_PerformSearch2821<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2822: CUDA_DES_PerformSearch2822<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2823: CUDA_DES_PerformSearch2823<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2824: CUDA_DES_PerformSearch2824<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2825: CUDA_DES_PerformSearch2825<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2826: CUDA_DES_PerformSearch2826<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2827: CUDA_DES_PerformSearch2827<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2828: CUDA_DES_PerformSearch2828<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2829: CUDA_DES_PerformSearch2829<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2830: CUDA_DES_PerformSearch2830<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2831: CUDA_DES_PerformSearch2831<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2832: CUDA_DES_PerformSearch2832<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2833: CUDA_DES_PerformSearch2833<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2834: CUDA_DES_PerformSearch2834<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2835: CUDA_DES_PerformSearch2835<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2836: CUDA_DES_PerformSearch2836<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2837: CUDA_DES_PerformSearch2837<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2838: CUDA_DES_PerformSearch2838<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2839: CUDA_DES_PerformSearch2839<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2840: CUDA_DES_PerformSearch2840<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2841: CUDA_DES_PerformSearch2841<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2842: CUDA_DES_PerformSearch2842<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2843: CUDA_DES_PerformSearch2843<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2844: CUDA_DES_PerformSearch2844<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2845: CUDA_DES_PerformSearch2845<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2846: CUDA_DES_PerformSearch2846<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2847: CUDA_DES_PerformSearch2847<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2848: CUDA_DES_PerformSearch2848<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2849: CUDA_DES_PerformSearch2849<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2850: CUDA_DES_PerformSearch2850<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2851: CUDA_DES_PerformSearch2851<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2852: CUDA_DES_PerformSearch2852<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2853: CUDA_DES_PerformSearch2853<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2854: CUDA_DES_PerformSearch2854<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2855: CUDA_DES_PerformSearch2855<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2856: CUDA_DES_PerformSearch2856<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2857: CUDA_DES_PerformSearch2857<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2858: CUDA_DES_PerformSearch2858<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2859: CUDA_DES_PerformSearch2859<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2860: CUDA_DES_PerformSearch2860<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2861: CUDA_DES_PerformSearch2861<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2862: CUDA_DES_PerformSearch2862<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2863: CUDA_DES_PerformSearch2863<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2864: CUDA_DES_PerformSearch2864<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2865: CUDA_DES_PerformSearch2865<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2866: CUDA_DES_PerformSearch2866<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2867: CUDA_DES_PerformSearch2867<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2868: CUDA_DES_PerformSearch2868<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2869: CUDA_DES_PerformSearch2869<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2870: CUDA_DES_PerformSearch2870<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2871: CUDA_DES_PerformSearch2871<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2872: CUDA_DES_PerformSearch2872<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2873: CUDA_DES_PerformSearch2873<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2874: CUDA_DES_PerformSearch2874<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2875: CUDA_DES_PerformSearch2875<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2876: CUDA_DES_PerformSearch2876<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2877: CUDA_DES_PerformSearch2877<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2878: CUDA_DES_PerformSearch2878<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2879: CUDA_DES_PerformSearch2879<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2880: CUDA_DES_PerformSearch2880<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2881: CUDA_DES_PerformSearch2881<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2882: CUDA_DES_PerformSearch2882<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2883: CUDA_DES_PerformSearch2883<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2884: CUDA_DES_PerformSearch2884<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2885: CUDA_DES_PerformSearch2885<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2886: CUDA_DES_PerformSearch2886<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2887: CUDA_DES_PerformSearch2887<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2888: CUDA_DES_PerformSearch2888<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2889: CUDA_DES_PerformSearch2889<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2890: CUDA_DES_PerformSearch2890<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2891: CUDA_DES_PerformSearch2891<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2892: CUDA_DES_PerformSearch2892<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2893: CUDA_DES_PerformSearch2893<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2894: CUDA_DES_PerformSearch2894<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2895: CUDA_DES_PerformSearch2895<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2896: CUDA_DES_PerformSearch2896<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2897: CUDA_DES_PerformSearch2897<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2898: CUDA_DES_PerformSearch2898<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2899: CUDA_DES_PerformSearch2899<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2900: CUDA_DES_PerformSearch2900<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2901: CUDA_DES_PerformSearch2901<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2902: CUDA_DES_PerformSearch2902<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2903: CUDA_DES_PerformSearch2903<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2904: CUDA_DES_PerformSearch2904<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2905: CUDA_DES_PerformSearch2905<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2906: CUDA_DES_PerformSearch2906<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2907: CUDA_DES_PerformSearch2907<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2908: CUDA_DES_PerformSearch2908<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2909: CUDA_DES_PerformSearch2909<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2910: CUDA_DES_PerformSearch2910<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2911: CUDA_DES_PerformSearch2911<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2912: CUDA_DES_PerformSearch2912<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2913: CUDA_DES_PerformSearch2913<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2914: CUDA_DES_PerformSearch2914<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2915: CUDA_DES_PerformSearch2915<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2916: CUDA_DES_PerformSearch2916<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2917: CUDA_DES_PerformSearch2917<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2918: CUDA_DES_PerformSearch2918<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2919: CUDA_DES_PerformSearch2919<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2920: CUDA_DES_PerformSearch2920<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2921: CUDA_DES_PerformSearch2921<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2922: CUDA_DES_PerformSearch2922<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2923: CUDA_DES_PerformSearch2923<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2924: CUDA_DES_PerformSearch2924<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2925: CUDA_DES_PerformSearch2925<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2926: CUDA_DES_PerformSearch2926<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2927: CUDA_DES_PerformSearch2927<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2928: CUDA_DES_PerformSearch2928<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2929: CUDA_DES_PerformSearch2929<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2930: CUDA_DES_PerformSearch2930<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2931: CUDA_DES_PerformSearch2931<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2932: CUDA_DES_PerformSearch2932<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2933: CUDA_DES_PerformSearch2933<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2934: CUDA_DES_PerformSearch2934<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2935: CUDA_DES_PerformSearch2935<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2936: CUDA_DES_PerformSearch2936<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2937: CUDA_DES_PerformSearch2937<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2938: CUDA_DES_PerformSearch2938<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2939: CUDA_DES_PerformSearch2939<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2940: CUDA_DES_PerformSearch2940<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2941: CUDA_DES_PerformSearch2941<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2942: CUDA_DES_PerformSearch2942<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2943: CUDA_DES_PerformSearch2943<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2944: CUDA_DES_PerformSearch2944<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2945: CUDA_DES_PerformSearch2945<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2946: CUDA_DES_PerformSearch2946<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2947: CUDA_DES_PerformSearch2947<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2948: CUDA_DES_PerformSearch2948<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2949: CUDA_DES_PerformSearch2949<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2950: CUDA_DES_PerformSearch2950<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2951: CUDA_DES_PerformSearch2951<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2952: CUDA_DES_PerformSearch2952<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2953: CUDA_DES_PerformSearch2953<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2954: CUDA_DES_PerformSearch2954<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2955: CUDA_DES_PerformSearch2955<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2956: CUDA_DES_PerformSearch2956<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2957: CUDA_DES_PerformSearch2957<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2958: CUDA_DES_PerformSearch2958<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2959: CUDA_DES_PerformSearch2959<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2960: CUDA_DES_PerformSearch2960<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2961: CUDA_DES_PerformSearch2961<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2962: CUDA_DES_PerformSearch2962<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2963: CUDA_DES_PerformSearch2963<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2964: CUDA_DES_PerformSearch2964<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2965: CUDA_DES_PerformSearch2965<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2966: CUDA_DES_PerformSearch2966<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2967: CUDA_DES_PerformSearch2967<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2968: CUDA_DES_PerformSearch2968<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2969: CUDA_DES_PerformSearch2969<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2970: CUDA_DES_PerformSearch2970<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2971: CUDA_DES_PerformSearch2971<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2972: CUDA_DES_PerformSearch2972<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2973: CUDA_DES_PerformSearch2973<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2974: CUDA_DES_PerformSearch2974<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2975: CUDA_DES_PerformSearch2975<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2976: CUDA_DES_PerformSearch2976<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2977: CUDA_DES_PerformSearch2977<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2978: CUDA_DES_PerformSearch2978<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2979: CUDA_DES_PerformSearch2979<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2980: CUDA_DES_PerformSearch2980<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2981: CUDA_DES_PerformSearch2981<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2982: CUDA_DES_PerformSearch2982<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2983: CUDA_DES_PerformSearch2983<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2984: CUDA_DES_PerformSearch2984<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2985: CUDA_DES_PerformSearch2985<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2986: CUDA_DES_PerformSearch2986<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2987: CUDA_DES_PerformSearch2987<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2988: CUDA_DES_PerformSearch2988<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2989: CUDA_DES_PerformSearch2989<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2990: CUDA_DES_PerformSearch2990<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2991: CUDA_DES_PerformSearch2991<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2992: CUDA_DES_PerformSearch2992<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2993: CUDA_DES_PerformSearch2993<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2994: CUDA_DES_PerformSearch2994<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2995: CUDA_DES_PerformSearch2995<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2996: CUDA_DES_PerformSearch2996<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2997: CUDA_DES_PerformSearch2997<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2998: CUDA_DES_PerformSearch2998<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2999: CUDA_DES_PerformSearch2999<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3000: CUDA_DES_PerformSearch3000<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3001: CUDA_DES_PerformSearch3001<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3002: CUDA_DES_PerformSearch3002<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3003: CUDA_DES_PerformSearch3003<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3004: CUDA_DES_PerformSearch3004<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3005: CUDA_DES_PerformSearch3005<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3006: CUDA_DES_PerformSearch3006<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3007: CUDA_DES_PerformSearch3007<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3008: CUDA_DES_PerformSearch3008<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3009: CUDA_DES_PerformSearch3009<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3010: CUDA_DES_PerformSearch3010<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3011: CUDA_DES_PerformSearch3011<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3012: CUDA_DES_PerformSearch3012<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3013: CUDA_DES_PerformSearch3013<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3014: CUDA_DES_PerformSearch3014<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3015: CUDA_DES_PerformSearch3015<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3016: CUDA_DES_PerformSearch3016<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3017: CUDA_DES_PerformSearch3017<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3018: CUDA_DES_PerformSearch3018<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3019: CUDA_DES_PerformSearch3019<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3020: CUDA_DES_PerformSearch3020<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3021: CUDA_DES_PerformSearch3021<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3022: CUDA_DES_PerformSearch3022<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3023: CUDA_DES_PerformSearch3023<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3024: CUDA_DES_PerformSearch3024<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3025: CUDA_DES_PerformSearch3025<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3026: CUDA_DES_PerformSearch3026<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3027: CUDA_DES_PerformSearch3027<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3028: CUDA_DES_PerformSearch3028<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3029: CUDA_DES_PerformSearch3029<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3030: CUDA_DES_PerformSearch3030<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3031: CUDA_DES_PerformSearch3031<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3032: CUDA_DES_PerformSearch3032<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3033: CUDA_DES_PerformSearch3033<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3034: CUDA_DES_PerformSearch3034<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3035: CUDA_DES_PerformSearch3035<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3036: CUDA_DES_PerformSearch3036<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3037: CUDA_DES_PerformSearch3037<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3038: CUDA_DES_PerformSearch3038<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3039: CUDA_DES_PerformSearch3039<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3040: CUDA_DES_PerformSearch3040<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3041: CUDA_DES_PerformSearch3041<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3042: CUDA_DES_PerformSearch3042<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3043: CUDA_DES_PerformSearch3043<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3044: CUDA_DES_PerformSearch3044<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3045: CUDA_DES_PerformSearch3045<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3046: CUDA_DES_PerformSearch3046<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3047: CUDA_DES_PerformSearch3047<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3048: CUDA_DES_PerformSearch3048<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3049: CUDA_DES_PerformSearch3049<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3050: CUDA_DES_PerformSearch3050<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3051: CUDA_DES_PerformSearch3051<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3052: CUDA_DES_PerformSearch3052<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3053: CUDA_DES_PerformSearch3053<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3054: CUDA_DES_PerformSearch3054<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3055: CUDA_DES_PerformSearch3055<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3056: CUDA_DES_PerformSearch3056<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3057: CUDA_DES_PerformSearch3057<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3058: CUDA_DES_PerformSearch3058<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3059: CUDA_DES_PerformSearch3059<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3060: CUDA_DES_PerformSearch3060<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3061: CUDA_DES_PerformSearch3061<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3062: CUDA_DES_PerformSearch3062<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3063: CUDA_DES_PerformSearch3063<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3064: CUDA_DES_PerformSearch3064<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3065: CUDA_DES_PerformSearch3065<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3066: CUDA_DES_PerformSearch3066<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3067: CUDA_DES_PerformSearch3067<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3068: CUDA_DES_PerformSearch3068<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3069: CUDA_DES_PerformSearch3069<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3070: CUDA_DES_PerformSearch3070<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 3071: CUDA_DES_PerformSearch3071<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
