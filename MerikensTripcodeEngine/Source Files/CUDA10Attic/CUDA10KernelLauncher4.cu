// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "CUDA10KernelCommon.h"

#define SALT 2048
#include "CUDA10Kernel.h"
#define SALT 2049
#include "CUDA10Kernel.h"
#define SALT 2050
#include "CUDA10Kernel.h"
#define SALT 2051
#include "CUDA10Kernel.h"
#define SALT 2052
#include "CUDA10Kernel.h"
#define SALT 2053
#include "CUDA10Kernel.h"
#define SALT 2054
#include "CUDA10Kernel.h"
#define SALT 2055
#include "CUDA10Kernel.h"
#define SALT 2056
#include "CUDA10Kernel.h"
#define SALT 2057
#include "CUDA10Kernel.h"
#define SALT 2058
#include "CUDA10Kernel.h"
#define SALT 2059
#include "CUDA10Kernel.h"
#define SALT 2060
#include "CUDA10Kernel.h"
#define SALT 2061
#include "CUDA10Kernel.h"
#define SALT 2062
#include "CUDA10Kernel.h"
#define SALT 2063
#include "CUDA10Kernel.h"
#define SALT 2064
#include "CUDA10Kernel.h"
#define SALT 2065
#include "CUDA10Kernel.h"
#define SALT 2066
#include "CUDA10Kernel.h"
#define SALT 2067
#include "CUDA10Kernel.h"
#define SALT 2068
#include "CUDA10Kernel.h"
#define SALT 2069
#include "CUDA10Kernel.h"
#define SALT 2070
#include "CUDA10Kernel.h"
#define SALT 2071
#include "CUDA10Kernel.h"
#define SALT 2072
#include "CUDA10Kernel.h"
#define SALT 2073
#include "CUDA10Kernel.h"
#define SALT 2074
#include "CUDA10Kernel.h"
#define SALT 2075
#include "CUDA10Kernel.h"
#define SALT 2076
#include "CUDA10Kernel.h"
#define SALT 2077
#include "CUDA10Kernel.h"
#define SALT 2078
#include "CUDA10Kernel.h"
#define SALT 2079
#include "CUDA10Kernel.h"
#define SALT 2080
#include "CUDA10Kernel.h"
#define SALT 2081
#include "CUDA10Kernel.h"
#define SALT 2082
#include "CUDA10Kernel.h"
#define SALT 2083
#include "CUDA10Kernel.h"
#define SALT 2084
#include "CUDA10Kernel.h"
#define SALT 2085
#include "CUDA10Kernel.h"
#define SALT 2086
#include "CUDA10Kernel.h"
#define SALT 2087
#include "CUDA10Kernel.h"
#define SALT 2088
#include "CUDA10Kernel.h"
#define SALT 2089
#include "CUDA10Kernel.h"
#define SALT 2090
#include "CUDA10Kernel.h"
#define SALT 2091
#include "CUDA10Kernel.h"
#define SALT 2092
#include "CUDA10Kernel.h"
#define SALT 2093
#include "CUDA10Kernel.h"
#define SALT 2094
#include "CUDA10Kernel.h"
#define SALT 2095
#include "CUDA10Kernel.h"
#define SALT 2096
#include "CUDA10Kernel.h"
#define SALT 2097
#include "CUDA10Kernel.h"
#define SALT 2098
#include "CUDA10Kernel.h"
#define SALT 2099
#include "CUDA10Kernel.h"
#define SALT 2100
#include "CUDA10Kernel.h"
#define SALT 2101
#include "CUDA10Kernel.h"
#define SALT 2102
#include "CUDA10Kernel.h"
#define SALT 2103
#include "CUDA10Kernel.h"
#define SALT 2104
#include "CUDA10Kernel.h"
#define SALT 2105
#include "CUDA10Kernel.h"
#define SALT 2106
#include "CUDA10Kernel.h"
#define SALT 2107
#include "CUDA10Kernel.h"
#define SALT 2108
#include "CUDA10Kernel.h"
#define SALT 2109
#include "CUDA10Kernel.h"
#define SALT 2110
#include "CUDA10Kernel.h"
#define SALT 2111
#include "CUDA10Kernel.h"
#define SALT 2112
#include "CUDA10Kernel.h"
#define SALT 2113
#include "CUDA10Kernel.h"
#define SALT 2114
#include "CUDA10Kernel.h"
#define SALT 2115
#include "CUDA10Kernel.h"
#define SALT 2116
#include "CUDA10Kernel.h"
#define SALT 2117
#include "CUDA10Kernel.h"
#define SALT 2118
#include "CUDA10Kernel.h"
#define SALT 2119
#include "CUDA10Kernel.h"
#define SALT 2120
#include "CUDA10Kernel.h"
#define SALT 2121
#include "CUDA10Kernel.h"
#define SALT 2122
#include "CUDA10Kernel.h"
#define SALT 2123
#include "CUDA10Kernel.h"
#define SALT 2124
#include "CUDA10Kernel.h"
#define SALT 2125
#include "CUDA10Kernel.h"
#define SALT 2126
#include "CUDA10Kernel.h"
#define SALT 2127
#include "CUDA10Kernel.h"
#define SALT 2128
#include "CUDA10Kernel.h"
#define SALT 2129
#include "CUDA10Kernel.h"
#define SALT 2130
#include "CUDA10Kernel.h"
#define SALT 2131
#include "CUDA10Kernel.h"
#define SALT 2132
#include "CUDA10Kernel.h"
#define SALT 2133
#include "CUDA10Kernel.h"
#define SALT 2134
#include "CUDA10Kernel.h"
#define SALT 2135
#include "CUDA10Kernel.h"
#define SALT 2136
#include "CUDA10Kernel.h"
#define SALT 2137
#include "CUDA10Kernel.h"
#define SALT 2138
#include "CUDA10Kernel.h"
#define SALT 2139
#include "CUDA10Kernel.h"
#define SALT 2140
#include "CUDA10Kernel.h"
#define SALT 2141
#include "CUDA10Kernel.h"
#define SALT 2142
#include "CUDA10Kernel.h"
#define SALT 2143
#include "CUDA10Kernel.h"
#define SALT 2144
#include "CUDA10Kernel.h"
#define SALT 2145
#include "CUDA10Kernel.h"
#define SALT 2146
#include "CUDA10Kernel.h"
#define SALT 2147
#include "CUDA10Kernel.h"
#define SALT 2148
#include "CUDA10Kernel.h"
#define SALT 2149
#include "CUDA10Kernel.h"
#define SALT 2150
#include "CUDA10Kernel.h"
#define SALT 2151
#include "CUDA10Kernel.h"
#define SALT 2152
#include "CUDA10Kernel.h"
#define SALT 2153
#include "CUDA10Kernel.h"
#define SALT 2154
#include "CUDA10Kernel.h"
#define SALT 2155
#include "CUDA10Kernel.h"
#define SALT 2156
#include "CUDA10Kernel.h"
#define SALT 2157
#include "CUDA10Kernel.h"
#define SALT 2158
#include "CUDA10Kernel.h"
#define SALT 2159
#include "CUDA10Kernel.h"
#define SALT 2160
#include "CUDA10Kernel.h"
#define SALT 2161
#include "CUDA10Kernel.h"
#define SALT 2162
#include "CUDA10Kernel.h"
#define SALT 2163
#include "CUDA10Kernel.h"
#define SALT 2164
#include "CUDA10Kernel.h"
#define SALT 2165
#include "CUDA10Kernel.h"
#define SALT 2166
#include "CUDA10Kernel.h"
#define SALT 2167
#include "CUDA10Kernel.h"
#define SALT 2168
#include "CUDA10Kernel.h"
#define SALT 2169
#include "CUDA10Kernel.h"
#define SALT 2170
#include "CUDA10Kernel.h"
#define SALT 2171
#include "CUDA10Kernel.h"
#define SALT 2172
#include "CUDA10Kernel.h"
#define SALT 2173
#include "CUDA10Kernel.h"
#define SALT 2174
#include "CUDA10Kernel.h"
#define SALT 2175
#include "CUDA10Kernel.h"
#define SALT 2176
#include "CUDA10Kernel.h"
#define SALT 2177
#include "CUDA10Kernel.h"
#define SALT 2178
#include "CUDA10Kernel.h"
#define SALT 2179
#include "CUDA10Kernel.h"
#define SALT 2180
#include "CUDA10Kernel.h"
#define SALT 2181
#include "CUDA10Kernel.h"
#define SALT 2182
#include "CUDA10Kernel.h"
#define SALT 2183
#include "CUDA10Kernel.h"
#define SALT 2184
#include "CUDA10Kernel.h"
#define SALT 2185
#include "CUDA10Kernel.h"
#define SALT 2186
#include "CUDA10Kernel.h"
#define SALT 2187
#include "CUDA10Kernel.h"
#define SALT 2188
#include "CUDA10Kernel.h"
#define SALT 2189
#include "CUDA10Kernel.h"
#define SALT 2190
#include "CUDA10Kernel.h"
#define SALT 2191
#include "CUDA10Kernel.h"
#define SALT 2192
#include "CUDA10Kernel.h"
#define SALT 2193
#include "CUDA10Kernel.h"
#define SALT 2194
#include "CUDA10Kernel.h"
#define SALT 2195
#include "CUDA10Kernel.h"
#define SALT 2196
#include "CUDA10Kernel.h"
#define SALT 2197
#include "CUDA10Kernel.h"
#define SALT 2198
#include "CUDA10Kernel.h"
#define SALT 2199
#include "CUDA10Kernel.h"
#define SALT 2200
#include "CUDA10Kernel.h"
#define SALT 2201
#include "CUDA10Kernel.h"
#define SALT 2202
#include "CUDA10Kernel.h"
#define SALT 2203
#include "CUDA10Kernel.h"
#define SALT 2204
#include "CUDA10Kernel.h"
#define SALT 2205
#include "CUDA10Kernel.h"
#define SALT 2206
#include "CUDA10Kernel.h"
#define SALT 2207
#include "CUDA10Kernel.h"
#define SALT 2208
#include "CUDA10Kernel.h"
#define SALT 2209
#include "CUDA10Kernel.h"
#define SALT 2210
#include "CUDA10Kernel.h"
#define SALT 2211
#include "CUDA10Kernel.h"
#define SALT 2212
#include "CUDA10Kernel.h"
#define SALT 2213
#include "CUDA10Kernel.h"
#define SALT 2214
#include "CUDA10Kernel.h"
#define SALT 2215
#include "CUDA10Kernel.h"
#define SALT 2216
#include "CUDA10Kernel.h"
#define SALT 2217
#include "CUDA10Kernel.h"
#define SALT 2218
#include "CUDA10Kernel.h"
#define SALT 2219
#include "CUDA10Kernel.h"
#define SALT 2220
#include "CUDA10Kernel.h"
#define SALT 2221
#include "CUDA10Kernel.h"
#define SALT 2222
#include "CUDA10Kernel.h"
#define SALT 2223
#include "CUDA10Kernel.h"
#define SALT 2224
#include "CUDA10Kernel.h"
#define SALT 2225
#include "CUDA10Kernel.h"
#define SALT 2226
#include "CUDA10Kernel.h"
#define SALT 2227
#include "CUDA10Kernel.h"
#define SALT 2228
#include "CUDA10Kernel.h"
#define SALT 2229
#include "CUDA10Kernel.h"
#define SALT 2230
#include "CUDA10Kernel.h"
#define SALT 2231
#include "CUDA10Kernel.h"
#define SALT 2232
#include "CUDA10Kernel.h"
#define SALT 2233
#include "CUDA10Kernel.h"
#define SALT 2234
#include "CUDA10Kernel.h"
#define SALT 2235
#include "CUDA10Kernel.h"
#define SALT 2236
#include "CUDA10Kernel.h"
#define SALT 2237
#include "CUDA10Kernel.h"
#define SALT 2238
#include "CUDA10Kernel.h"
#define SALT 2239
#include "CUDA10Kernel.h"
#define SALT 2240
#include "CUDA10Kernel.h"
#define SALT 2241
#include "CUDA10Kernel.h"
#define SALT 2242
#include "CUDA10Kernel.h"
#define SALT 2243
#include "CUDA10Kernel.h"
#define SALT 2244
#include "CUDA10Kernel.h"
#define SALT 2245
#include "CUDA10Kernel.h"
#define SALT 2246
#include "CUDA10Kernel.h"
#define SALT 2247
#include "CUDA10Kernel.h"
#define SALT 2248
#include "CUDA10Kernel.h"
#define SALT 2249
#include "CUDA10Kernel.h"
#define SALT 2250
#include "CUDA10Kernel.h"
#define SALT 2251
#include "CUDA10Kernel.h"
#define SALT 2252
#include "CUDA10Kernel.h"
#define SALT 2253
#include "CUDA10Kernel.h"
#define SALT 2254
#include "CUDA10Kernel.h"
#define SALT 2255
#include "CUDA10Kernel.h"
#define SALT 2256
#include "CUDA10Kernel.h"
#define SALT 2257
#include "CUDA10Kernel.h"
#define SALT 2258
#include "CUDA10Kernel.h"
#define SALT 2259
#include "CUDA10Kernel.h"
#define SALT 2260
#include "CUDA10Kernel.h"
#define SALT 2261
#include "CUDA10Kernel.h"
#define SALT 2262
#include "CUDA10Kernel.h"
#define SALT 2263
#include "CUDA10Kernel.h"
#define SALT 2264
#include "CUDA10Kernel.h"
#define SALT 2265
#include "CUDA10Kernel.h"
#define SALT 2266
#include "CUDA10Kernel.h"
#define SALT 2267
#include "CUDA10Kernel.h"
#define SALT 2268
#include "CUDA10Kernel.h"
#define SALT 2269
#include "CUDA10Kernel.h"
#define SALT 2270
#include "CUDA10Kernel.h"
#define SALT 2271
#include "CUDA10Kernel.h"
#define SALT 2272
#include "CUDA10Kernel.h"
#define SALT 2273
#include "CUDA10Kernel.h"
#define SALT 2274
#include "CUDA10Kernel.h"
#define SALT 2275
#include "CUDA10Kernel.h"
#define SALT 2276
#include "CUDA10Kernel.h"
#define SALT 2277
#include "CUDA10Kernel.h"
#define SALT 2278
#include "CUDA10Kernel.h"
#define SALT 2279
#include "CUDA10Kernel.h"
#define SALT 2280
#include "CUDA10Kernel.h"
#define SALT 2281
#include "CUDA10Kernel.h"
#define SALT 2282
#include "CUDA10Kernel.h"
#define SALT 2283
#include "CUDA10Kernel.h"
#define SALT 2284
#include "CUDA10Kernel.h"
#define SALT 2285
#include "CUDA10Kernel.h"
#define SALT 2286
#include "CUDA10Kernel.h"
#define SALT 2287
#include "CUDA10Kernel.h"
#define SALT 2288
#include "CUDA10Kernel.h"
#define SALT 2289
#include "CUDA10Kernel.h"
#define SALT 2290
#include "CUDA10Kernel.h"
#define SALT 2291
#include "CUDA10Kernel.h"
#define SALT 2292
#include "CUDA10Kernel.h"
#define SALT 2293
#include "CUDA10Kernel.h"
#define SALT 2294
#include "CUDA10Kernel.h"
#define SALT 2295
#include "CUDA10Kernel.h"
#define SALT 2296
#include "CUDA10Kernel.h"
#define SALT 2297
#include "CUDA10Kernel.h"
#define SALT 2298
#include "CUDA10Kernel.h"
#define SALT 2299
#include "CUDA10Kernel.h"
#define SALT 2300
#include "CUDA10Kernel.h"
#define SALT 2301
#include "CUDA10Kernel.h"
#define SALT 2302
#include "CUDA10Kernel.h"
#define SALT 2303
#include "CUDA10Kernel.h"
#define SALT 2304
#include "CUDA10Kernel.h"
#define SALT 2305
#include "CUDA10Kernel.h"
#define SALT 2306
#include "CUDA10Kernel.h"
#define SALT 2307
#include "CUDA10Kernel.h"
#define SALT 2308
#include "CUDA10Kernel.h"
#define SALT 2309
#include "CUDA10Kernel.h"
#define SALT 2310
#include "CUDA10Kernel.h"
#define SALT 2311
#include "CUDA10Kernel.h"
#define SALT 2312
#include "CUDA10Kernel.h"
#define SALT 2313
#include "CUDA10Kernel.h"
#define SALT 2314
#include "CUDA10Kernel.h"
#define SALT 2315
#include "CUDA10Kernel.h"
#define SALT 2316
#include "CUDA10Kernel.h"
#define SALT 2317
#include "CUDA10Kernel.h"
#define SALT 2318
#include "CUDA10Kernel.h"
#define SALT 2319
#include "CUDA10Kernel.h"
#define SALT 2320
#include "CUDA10Kernel.h"
#define SALT 2321
#include "CUDA10Kernel.h"
#define SALT 2322
#include "CUDA10Kernel.h"
#define SALT 2323
#include "CUDA10Kernel.h"
#define SALT 2324
#include "CUDA10Kernel.h"
#define SALT 2325
#include "CUDA10Kernel.h"
#define SALT 2326
#include "CUDA10Kernel.h"
#define SALT 2327
#include "CUDA10Kernel.h"
#define SALT 2328
#include "CUDA10Kernel.h"
#define SALT 2329
#include "CUDA10Kernel.h"
#define SALT 2330
#include "CUDA10Kernel.h"
#define SALT 2331
#include "CUDA10Kernel.h"
#define SALT 2332
#include "CUDA10Kernel.h"
#define SALT 2333
#include "CUDA10Kernel.h"
#define SALT 2334
#include "CUDA10Kernel.h"
#define SALT 2335
#include "CUDA10Kernel.h"
#define SALT 2336
#include "CUDA10Kernel.h"
#define SALT 2337
#include "CUDA10Kernel.h"
#define SALT 2338
#include "CUDA10Kernel.h"
#define SALT 2339
#include "CUDA10Kernel.h"
#define SALT 2340
#include "CUDA10Kernel.h"
#define SALT 2341
#include "CUDA10Kernel.h"
#define SALT 2342
#include "CUDA10Kernel.h"
#define SALT 2343
#include "CUDA10Kernel.h"
#define SALT 2344
#include "CUDA10Kernel.h"
#define SALT 2345
#include "CUDA10Kernel.h"
#define SALT 2346
#include "CUDA10Kernel.h"
#define SALT 2347
#include "CUDA10Kernel.h"
#define SALT 2348
#include "CUDA10Kernel.h"
#define SALT 2349
#include "CUDA10Kernel.h"
#define SALT 2350
#include "CUDA10Kernel.h"
#define SALT 2351
#include "CUDA10Kernel.h"
#define SALT 2352
#include "CUDA10Kernel.h"
#define SALT 2353
#include "CUDA10Kernel.h"
#define SALT 2354
#include "CUDA10Kernel.h"
#define SALT 2355
#include "CUDA10Kernel.h"
#define SALT 2356
#include "CUDA10Kernel.h"
#define SALT 2357
#include "CUDA10Kernel.h"
#define SALT 2358
#include "CUDA10Kernel.h"
#define SALT 2359
#include "CUDA10Kernel.h"
#define SALT 2360
#include "CUDA10Kernel.h"
#define SALT 2361
#include "CUDA10Kernel.h"
#define SALT 2362
#include "CUDA10Kernel.h"
#define SALT 2363
#include "CUDA10Kernel.h"
#define SALT 2364
#include "CUDA10Kernel.h"
#define SALT 2365
#include "CUDA10Kernel.h"
#define SALT 2366
#include "CUDA10Kernel.h"
#define SALT 2367
#include "CUDA10Kernel.h"
#define SALT 2368
#include "CUDA10Kernel.h"
#define SALT 2369
#include "CUDA10Kernel.h"
#define SALT 2370
#include "CUDA10Kernel.h"
#define SALT 2371
#include "CUDA10Kernel.h"
#define SALT 2372
#include "CUDA10Kernel.h"
#define SALT 2373
#include "CUDA10Kernel.h"
#define SALT 2374
#include "CUDA10Kernel.h"
#define SALT 2375
#include "CUDA10Kernel.h"
#define SALT 2376
#include "CUDA10Kernel.h"
#define SALT 2377
#include "CUDA10Kernel.h"
#define SALT 2378
#include "CUDA10Kernel.h"
#define SALT 2379
#include "CUDA10Kernel.h"
#define SALT 2380
#include "CUDA10Kernel.h"
#define SALT 2381
#include "CUDA10Kernel.h"
#define SALT 2382
#include "CUDA10Kernel.h"
#define SALT 2383
#include "CUDA10Kernel.h"
#define SALT 2384
#include "CUDA10Kernel.h"
#define SALT 2385
#include "CUDA10Kernel.h"
#define SALT 2386
#include "CUDA10Kernel.h"
#define SALT 2387
#include "CUDA10Kernel.h"
#define SALT 2388
#include "CUDA10Kernel.h"
#define SALT 2389
#include "CUDA10Kernel.h"
#define SALT 2390
#include "CUDA10Kernel.h"
#define SALT 2391
#include "CUDA10Kernel.h"
#define SALT 2392
#include "CUDA10Kernel.h"
#define SALT 2393
#include "CUDA10Kernel.h"
#define SALT 2394
#include "CUDA10Kernel.h"
#define SALT 2395
#include "CUDA10Kernel.h"
#define SALT 2396
#include "CUDA10Kernel.h"
#define SALT 2397
#include "CUDA10Kernel.h"
#define SALT 2398
#include "CUDA10Kernel.h"
#define SALT 2399
#include "CUDA10Kernel.h"
#define SALT 2400
#include "CUDA10Kernel.h"
#define SALT 2401
#include "CUDA10Kernel.h"
#define SALT 2402
#include "CUDA10Kernel.h"
#define SALT 2403
#include "CUDA10Kernel.h"
#define SALT 2404
#include "CUDA10Kernel.h"
#define SALT 2405
#include "CUDA10Kernel.h"
#define SALT 2406
#include "CUDA10Kernel.h"
#define SALT 2407
#include "CUDA10Kernel.h"
#define SALT 2408
#include "CUDA10Kernel.h"
#define SALT 2409
#include "CUDA10Kernel.h"
#define SALT 2410
#include "CUDA10Kernel.h"
#define SALT 2411
#include "CUDA10Kernel.h"
#define SALT 2412
#include "CUDA10Kernel.h"
#define SALT 2413
#include "CUDA10Kernel.h"
#define SALT 2414
#include "CUDA10Kernel.h"
#define SALT 2415
#include "CUDA10Kernel.h"
#define SALT 2416
#include "CUDA10Kernel.h"
#define SALT 2417
#include "CUDA10Kernel.h"
#define SALT 2418
#include "CUDA10Kernel.h"
#define SALT 2419
#include "CUDA10Kernel.h"
#define SALT 2420
#include "CUDA10Kernel.h"
#define SALT 2421
#include "CUDA10Kernel.h"
#define SALT 2422
#include "CUDA10Kernel.h"
#define SALT 2423
#include "CUDA10Kernel.h"
#define SALT 2424
#include "CUDA10Kernel.h"
#define SALT 2425
#include "CUDA10Kernel.h"
#define SALT 2426
#include "CUDA10Kernel.h"
#define SALT 2427
#include "CUDA10Kernel.h"
#define SALT 2428
#include "CUDA10Kernel.h"
#define SALT 2429
#include "CUDA10Kernel.h"
#define SALT 2430
#include "CUDA10Kernel.h"
#define SALT 2431
#include "CUDA10Kernel.h"
#define SALT 2432
#include "CUDA10Kernel.h"
#define SALT 2433
#include "CUDA10Kernel.h"
#define SALT 2434
#include "CUDA10Kernel.h"
#define SALT 2435
#include "CUDA10Kernel.h"
#define SALT 2436
#include "CUDA10Kernel.h"
#define SALT 2437
#include "CUDA10Kernel.h"
#define SALT 2438
#include "CUDA10Kernel.h"
#define SALT 2439
#include "CUDA10Kernel.h"
#define SALT 2440
#include "CUDA10Kernel.h"
#define SALT 2441
#include "CUDA10Kernel.h"
#define SALT 2442
#include "CUDA10Kernel.h"
#define SALT 2443
#include "CUDA10Kernel.h"
#define SALT 2444
#include "CUDA10Kernel.h"
#define SALT 2445
#include "CUDA10Kernel.h"
#define SALT 2446
#include "CUDA10Kernel.h"
#define SALT 2447
#include "CUDA10Kernel.h"
#define SALT 2448
#include "CUDA10Kernel.h"
#define SALT 2449
#include "CUDA10Kernel.h"
#define SALT 2450
#include "CUDA10Kernel.h"
#define SALT 2451
#include "CUDA10Kernel.h"
#define SALT 2452
#include "CUDA10Kernel.h"
#define SALT 2453
#include "CUDA10Kernel.h"
#define SALT 2454
#include "CUDA10Kernel.h"
#define SALT 2455
#include "CUDA10Kernel.h"
#define SALT 2456
#include "CUDA10Kernel.h"
#define SALT 2457
#include "CUDA10Kernel.h"
#define SALT 2458
#include "CUDA10Kernel.h"
#define SALT 2459
#include "CUDA10Kernel.h"
#define SALT 2460
#include "CUDA10Kernel.h"
#define SALT 2461
#include "CUDA10Kernel.h"
#define SALT 2462
#include "CUDA10Kernel.h"
#define SALT 2463
#include "CUDA10Kernel.h"
#define SALT 2464
#include "CUDA10Kernel.h"
#define SALT 2465
#include "CUDA10Kernel.h"
#define SALT 2466
#include "CUDA10Kernel.h"
#define SALT 2467
#include "CUDA10Kernel.h"
#define SALT 2468
#include "CUDA10Kernel.h"
#define SALT 2469
#include "CUDA10Kernel.h"
#define SALT 2470
#include "CUDA10Kernel.h"
#define SALT 2471
#include "CUDA10Kernel.h"
#define SALT 2472
#include "CUDA10Kernel.h"
#define SALT 2473
#include "CUDA10Kernel.h"
#define SALT 2474
#include "CUDA10Kernel.h"
#define SALT 2475
#include "CUDA10Kernel.h"
#define SALT 2476
#include "CUDA10Kernel.h"
#define SALT 2477
#include "CUDA10Kernel.h"
#define SALT 2478
#include "CUDA10Kernel.h"
#define SALT 2479
#include "CUDA10Kernel.h"
#define SALT 2480
#include "CUDA10Kernel.h"
#define SALT 2481
#include "CUDA10Kernel.h"
#define SALT 2482
#include "CUDA10Kernel.h"
#define SALT 2483
#include "CUDA10Kernel.h"
#define SALT 2484
#include "CUDA10Kernel.h"
#define SALT 2485
#include "CUDA10Kernel.h"
#define SALT 2486
#include "CUDA10Kernel.h"
#define SALT 2487
#include "CUDA10Kernel.h"
#define SALT 2488
#include "CUDA10Kernel.h"
#define SALT 2489
#include "CUDA10Kernel.h"
#define SALT 2490
#include "CUDA10Kernel.h"
#define SALT 2491
#include "CUDA10Kernel.h"
#define SALT 2492
#include "CUDA10Kernel.h"
#define SALT 2493
#include "CUDA10Kernel.h"
#define SALT 2494
#include "CUDA10Kernel.h"
#define SALT 2495
#include "CUDA10Kernel.h"
#define SALT 2496
#include "CUDA10Kernel.h"
#define SALT 2497
#include "CUDA10Kernel.h"
#define SALT 2498
#include "CUDA10Kernel.h"
#define SALT 2499
#include "CUDA10Kernel.h"
#define SALT 2500
#include "CUDA10Kernel.h"
#define SALT 2501
#include "CUDA10Kernel.h"
#define SALT 2502
#include "CUDA10Kernel.h"
#define SALT 2503
#include "CUDA10Kernel.h"
#define SALT 2504
#include "CUDA10Kernel.h"
#define SALT 2505
#include "CUDA10Kernel.h"
#define SALT 2506
#include "CUDA10Kernel.h"
#define SALT 2507
#include "CUDA10Kernel.h"
#define SALT 2508
#include "CUDA10Kernel.h"
#define SALT 2509
#include "CUDA10Kernel.h"
#define SALT 2510
#include "CUDA10Kernel.h"
#define SALT 2511
#include "CUDA10Kernel.h"
#define SALT 2512
#include "CUDA10Kernel.h"
#define SALT 2513
#include "CUDA10Kernel.h"
#define SALT 2514
#include "CUDA10Kernel.h"
#define SALT 2515
#include "CUDA10Kernel.h"
#define SALT 2516
#include "CUDA10Kernel.h"
#define SALT 2517
#include "CUDA10Kernel.h"
#define SALT 2518
#include "CUDA10Kernel.h"
#define SALT 2519
#include "CUDA10Kernel.h"
#define SALT 2520
#include "CUDA10Kernel.h"
#define SALT 2521
#include "CUDA10Kernel.h"
#define SALT 2522
#include "CUDA10Kernel.h"
#define SALT 2523
#include "CUDA10Kernel.h"
#define SALT 2524
#include "CUDA10Kernel.h"
#define SALT 2525
#include "CUDA10Kernel.h"
#define SALT 2526
#include "CUDA10Kernel.h"
#define SALT 2527
#include "CUDA10Kernel.h"
#define SALT 2528
#include "CUDA10Kernel.h"
#define SALT 2529
#include "CUDA10Kernel.h"
#define SALT 2530
#include "CUDA10Kernel.h"
#define SALT 2531
#include "CUDA10Kernel.h"
#define SALT 2532
#include "CUDA10Kernel.h"
#define SALT 2533
#include "CUDA10Kernel.h"
#define SALT 2534
#include "CUDA10Kernel.h"
#define SALT 2535
#include "CUDA10Kernel.h"
#define SALT 2536
#include "CUDA10Kernel.h"
#define SALT 2537
#include "CUDA10Kernel.h"
#define SALT 2538
#include "CUDA10Kernel.h"
#define SALT 2539
#include "CUDA10Kernel.h"
#define SALT 2540
#include "CUDA10Kernel.h"
#define SALT 2541
#include "CUDA10Kernel.h"
#define SALT 2542
#include "CUDA10Kernel.h"
#define SALT 2543
#include "CUDA10Kernel.h"
#define SALT 2544
#include "CUDA10Kernel.h"
#define SALT 2545
#include "CUDA10Kernel.h"
#define SALT 2546
#include "CUDA10Kernel.h"
#define SALT 2547
#include "CUDA10Kernel.h"
#define SALT 2548
#include "CUDA10Kernel.h"
#define SALT 2549
#include "CUDA10Kernel.h"
#define SALT 2550
#include "CUDA10Kernel.h"
#define SALT 2551
#include "CUDA10Kernel.h"
#define SALT 2552
#include "CUDA10Kernel.h"
#define SALT 2553
#include "CUDA10Kernel.h"
#define SALT 2554
#include "CUDA10Kernel.h"
#define SALT 2555
#include "CUDA10Kernel.h"
#define SALT 2556
#include "CUDA10Kernel.h"
#define SALT 2557
#include "CUDA10Kernel.h"
#define SALT 2558
#include "CUDA10Kernel.h"
#define SALT 2559
#include "CUDA10Kernel.h"



void CUDA_DES_InitializeKernelLauncher4()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel4(CUDA_DES_LAUNCH_KERNEL_ARGS)
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcodeKey));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(unsigned char) * 96));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key7Array),         key7Array,         sizeof(unsigned char) * CUDA_DES_BS_DEPTH));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyFrom49To55Array), keyFrom49To55Array, sizeof(DES_Vector) * 7));

	switch (intSalt) {
	case 2048: CUDA_DES_PerformSearch2048<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2049: CUDA_DES_PerformSearch2049<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2050: CUDA_DES_PerformSearch2050<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2051: CUDA_DES_PerformSearch2051<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2052: CUDA_DES_PerformSearch2052<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2053: CUDA_DES_PerformSearch2053<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2054: CUDA_DES_PerformSearch2054<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2055: CUDA_DES_PerformSearch2055<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2056: CUDA_DES_PerformSearch2056<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2057: CUDA_DES_PerformSearch2057<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2058: CUDA_DES_PerformSearch2058<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2059: CUDA_DES_PerformSearch2059<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2060: CUDA_DES_PerformSearch2060<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2061: CUDA_DES_PerformSearch2061<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2062: CUDA_DES_PerformSearch2062<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2063: CUDA_DES_PerformSearch2063<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2064: CUDA_DES_PerformSearch2064<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2065: CUDA_DES_PerformSearch2065<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2066: CUDA_DES_PerformSearch2066<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2067: CUDA_DES_PerformSearch2067<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2068: CUDA_DES_PerformSearch2068<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2069: CUDA_DES_PerformSearch2069<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2070: CUDA_DES_PerformSearch2070<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2071: CUDA_DES_PerformSearch2071<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2072: CUDA_DES_PerformSearch2072<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2073: CUDA_DES_PerformSearch2073<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2074: CUDA_DES_PerformSearch2074<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2075: CUDA_DES_PerformSearch2075<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2076: CUDA_DES_PerformSearch2076<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2077: CUDA_DES_PerformSearch2077<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2078: CUDA_DES_PerformSearch2078<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2079: CUDA_DES_PerformSearch2079<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2080: CUDA_DES_PerformSearch2080<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2081: CUDA_DES_PerformSearch2081<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2082: CUDA_DES_PerformSearch2082<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2083: CUDA_DES_PerformSearch2083<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2084: CUDA_DES_PerformSearch2084<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2085: CUDA_DES_PerformSearch2085<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2086: CUDA_DES_PerformSearch2086<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2087: CUDA_DES_PerformSearch2087<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2088: CUDA_DES_PerformSearch2088<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2089: CUDA_DES_PerformSearch2089<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2090: CUDA_DES_PerformSearch2090<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2091: CUDA_DES_PerformSearch2091<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2092: CUDA_DES_PerformSearch2092<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2093: CUDA_DES_PerformSearch2093<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2094: CUDA_DES_PerformSearch2094<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2095: CUDA_DES_PerformSearch2095<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2096: CUDA_DES_PerformSearch2096<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2097: CUDA_DES_PerformSearch2097<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2098: CUDA_DES_PerformSearch2098<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2099: CUDA_DES_PerformSearch2099<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2100: CUDA_DES_PerformSearch2100<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2101: CUDA_DES_PerformSearch2101<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2102: CUDA_DES_PerformSearch2102<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2103: CUDA_DES_PerformSearch2103<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2104: CUDA_DES_PerformSearch2104<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2105: CUDA_DES_PerformSearch2105<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2106: CUDA_DES_PerformSearch2106<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2107: CUDA_DES_PerformSearch2107<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2108: CUDA_DES_PerformSearch2108<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2109: CUDA_DES_PerformSearch2109<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2110: CUDA_DES_PerformSearch2110<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2111: CUDA_DES_PerformSearch2111<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2112: CUDA_DES_PerformSearch2112<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2113: CUDA_DES_PerformSearch2113<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2114: CUDA_DES_PerformSearch2114<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2115: CUDA_DES_PerformSearch2115<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2116: CUDA_DES_PerformSearch2116<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2117: CUDA_DES_PerformSearch2117<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2118: CUDA_DES_PerformSearch2118<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2119: CUDA_DES_PerformSearch2119<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2120: CUDA_DES_PerformSearch2120<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2121: CUDA_DES_PerformSearch2121<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2122: CUDA_DES_PerformSearch2122<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2123: CUDA_DES_PerformSearch2123<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2124: CUDA_DES_PerformSearch2124<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2125: CUDA_DES_PerformSearch2125<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2126: CUDA_DES_PerformSearch2126<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2127: CUDA_DES_PerformSearch2127<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2128: CUDA_DES_PerformSearch2128<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2129: CUDA_DES_PerformSearch2129<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2130: CUDA_DES_PerformSearch2130<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2131: CUDA_DES_PerformSearch2131<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2132: CUDA_DES_PerformSearch2132<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2133: CUDA_DES_PerformSearch2133<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2134: CUDA_DES_PerformSearch2134<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2135: CUDA_DES_PerformSearch2135<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2136: CUDA_DES_PerformSearch2136<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2137: CUDA_DES_PerformSearch2137<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2138: CUDA_DES_PerformSearch2138<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2139: CUDA_DES_PerformSearch2139<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2140: CUDA_DES_PerformSearch2140<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2141: CUDA_DES_PerformSearch2141<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2142: CUDA_DES_PerformSearch2142<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2143: CUDA_DES_PerformSearch2143<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2144: CUDA_DES_PerformSearch2144<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2145: CUDA_DES_PerformSearch2145<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2146: CUDA_DES_PerformSearch2146<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2147: CUDA_DES_PerformSearch2147<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2148: CUDA_DES_PerformSearch2148<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2149: CUDA_DES_PerformSearch2149<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2150: CUDA_DES_PerformSearch2150<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2151: CUDA_DES_PerformSearch2151<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2152: CUDA_DES_PerformSearch2152<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2153: CUDA_DES_PerformSearch2153<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2154: CUDA_DES_PerformSearch2154<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2155: CUDA_DES_PerformSearch2155<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2156: CUDA_DES_PerformSearch2156<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2157: CUDA_DES_PerformSearch2157<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2158: CUDA_DES_PerformSearch2158<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2159: CUDA_DES_PerformSearch2159<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2160: CUDA_DES_PerformSearch2160<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2161: CUDA_DES_PerformSearch2161<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2162: CUDA_DES_PerformSearch2162<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2163: CUDA_DES_PerformSearch2163<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2164: CUDA_DES_PerformSearch2164<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2165: CUDA_DES_PerformSearch2165<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2166: CUDA_DES_PerformSearch2166<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2167: CUDA_DES_PerformSearch2167<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2168: CUDA_DES_PerformSearch2168<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2169: CUDA_DES_PerformSearch2169<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2170: CUDA_DES_PerformSearch2170<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2171: CUDA_DES_PerformSearch2171<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2172: CUDA_DES_PerformSearch2172<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2173: CUDA_DES_PerformSearch2173<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2174: CUDA_DES_PerformSearch2174<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2175: CUDA_DES_PerformSearch2175<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2176: CUDA_DES_PerformSearch2176<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2177: CUDA_DES_PerformSearch2177<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2178: CUDA_DES_PerformSearch2178<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2179: CUDA_DES_PerformSearch2179<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2180: CUDA_DES_PerformSearch2180<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2181: CUDA_DES_PerformSearch2181<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2182: CUDA_DES_PerformSearch2182<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2183: CUDA_DES_PerformSearch2183<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2184: CUDA_DES_PerformSearch2184<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2185: CUDA_DES_PerformSearch2185<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2186: CUDA_DES_PerformSearch2186<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2187: CUDA_DES_PerformSearch2187<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2188: CUDA_DES_PerformSearch2188<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2189: CUDA_DES_PerformSearch2189<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2190: CUDA_DES_PerformSearch2190<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2191: CUDA_DES_PerformSearch2191<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2192: CUDA_DES_PerformSearch2192<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2193: CUDA_DES_PerformSearch2193<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2194: CUDA_DES_PerformSearch2194<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2195: CUDA_DES_PerformSearch2195<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2196: CUDA_DES_PerformSearch2196<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2197: CUDA_DES_PerformSearch2197<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2198: CUDA_DES_PerformSearch2198<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2199: CUDA_DES_PerformSearch2199<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2200: CUDA_DES_PerformSearch2200<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2201: CUDA_DES_PerformSearch2201<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2202: CUDA_DES_PerformSearch2202<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2203: CUDA_DES_PerformSearch2203<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2204: CUDA_DES_PerformSearch2204<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2205: CUDA_DES_PerformSearch2205<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2206: CUDA_DES_PerformSearch2206<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2207: CUDA_DES_PerformSearch2207<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2208: CUDA_DES_PerformSearch2208<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2209: CUDA_DES_PerformSearch2209<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2210: CUDA_DES_PerformSearch2210<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2211: CUDA_DES_PerformSearch2211<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2212: CUDA_DES_PerformSearch2212<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2213: CUDA_DES_PerformSearch2213<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2214: CUDA_DES_PerformSearch2214<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2215: CUDA_DES_PerformSearch2215<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2216: CUDA_DES_PerformSearch2216<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2217: CUDA_DES_PerformSearch2217<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2218: CUDA_DES_PerformSearch2218<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2219: CUDA_DES_PerformSearch2219<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2220: CUDA_DES_PerformSearch2220<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2221: CUDA_DES_PerformSearch2221<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2222: CUDA_DES_PerformSearch2222<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2223: CUDA_DES_PerformSearch2223<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2224: CUDA_DES_PerformSearch2224<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2225: CUDA_DES_PerformSearch2225<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2226: CUDA_DES_PerformSearch2226<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2227: CUDA_DES_PerformSearch2227<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2228: CUDA_DES_PerformSearch2228<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2229: CUDA_DES_PerformSearch2229<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2230: CUDA_DES_PerformSearch2230<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2231: CUDA_DES_PerformSearch2231<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2232: CUDA_DES_PerformSearch2232<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2233: CUDA_DES_PerformSearch2233<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2234: CUDA_DES_PerformSearch2234<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2235: CUDA_DES_PerformSearch2235<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2236: CUDA_DES_PerformSearch2236<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2237: CUDA_DES_PerformSearch2237<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2238: CUDA_DES_PerformSearch2238<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2239: CUDA_DES_PerformSearch2239<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2240: CUDA_DES_PerformSearch2240<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2241: CUDA_DES_PerformSearch2241<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2242: CUDA_DES_PerformSearch2242<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2243: CUDA_DES_PerformSearch2243<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2244: CUDA_DES_PerformSearch2244<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2245: CUDA_DES_PerformSearch2245<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2246: CUDA_DES_PerformSearch2246<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2247: CUDA_DES_PerformSearch2247<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2248: CUDA_DES_PerformSearch2248<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2249: CUDA_DES_PerformSearch2249<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2250: CUDA_DES_PerformSearch2250<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2251: CUDA_DES_PerformSearch2251<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2252: CUDA_DES_PerformSearch2252<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2253: CUDA_DES_PerformSearch2253<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2254: CUDA_DES_PerformSearch2254<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2255: CUDA_DES_PerformSearch2255<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2256: CUDA_DES_PerformSearch2256<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2257: CUDA_DES_PerformSearch2257<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2258: CUDA_DES_PerformSearch2258<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2259: CUDA_DES_PerformSearch2259<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2260: CUDA_DES_PerformSearch2260<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2261: CUDA_DES_PerformSearch2261<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2262: CUDA_DES_PerformSearch2262<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2263: CUDA_DES_PerformSearch2263<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2264: CUDA_DES_PerformSearch2264<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2265: CUDA_DES_PerformSearch2265<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2266: CUDA_DES_PerformSearch2266<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2267: CUDA_DES_PerformSearch2267<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2268: CUDA_DES_PerformSearch2268<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2269: CUDA_DES_PerformSearch2269<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2270: CUDA_DES_PerformSearch2270<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2271: CUDA_DES_PerformSearch2271<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2272: CUDA_DES_PerformSearch2272<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2273: CUDA_DES_PerformSearch2273<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2274: CUDA_DES_PerformSearch2274<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2275: CUDA_DES_PerformSearch2275<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2276: CUDA_DES_PerformSearch2276<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2277: CUDA_DES_PerformSearch2277<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2278: CUDA_DES_PerformSearch2278<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2279: CUDA_DES_PerformSearch2279<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2280: CUDA_DES_PerformSearch2280<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2281: CUDA_DES_PerformSearch2281<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2282: CUDA_DES_PerformSearch2282<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2283: CUDA_DES_PerformSearch2283<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2284: CUDA_DES_PerformSearch2284<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2285: CUDA_DES_PerformSearch2285<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2286: CUDA_DES_PerformSearch2286<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2287: CUDA_DES_PerformSearch2287<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2288: CUDA_DES_PerformSearch2288<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2289: CUDA_DES_PerformSearch2289<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2290: CUDA_DES_PerformSearch2290<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2291: CUDA_DES_PerformSearch2291<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2292: CUDA_DES_PerformSearch2292<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2293: CUDA_DES_PerformSearch2293<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2294: CUDA_DES_PerformSearch2294<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2295: CUDA_DES_PerformSearch2295<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2296: CUDA_DES_PerformSearch2296<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2297: CUDA_DES_PerformSearch2297<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2298: CUDA_DES_PerformSearch2298<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2299: CUDA_DES_PerformSearch2299<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2300: CUDA_DES_PerformSearch2300<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2301: CUDA_DES_PerformSearch2301<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2302: CUDA_DES_PerformSearch2302<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2303: CUDA_DES_PerformSearch2303<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2304: CUDA_DES_PerformSearch2304<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2305: CUDA_DES_PerformSearch2305<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2306: CUDA_DES_PerformSearch2306<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2307: CUDA_DES_PerformSearch2307<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2308: CUDA_DES_PerformSearch2308<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2309: CUDA_DES_PerformSearch2309<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2310: CUDA_DES_PerformSearch2310<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2311: CUDA_DES_PerformSearch2311<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2312: CUDA_DES_PerformSearch2312<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2313: CUDA_DES_PerformSearch2313<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2314: CUDA_DES_PerformSearch2314<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2315: CUDA_DES_PerformSearch2315<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2316: CUDA_DES_PerformSearch2316<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2317: CUDA_DES_PerformSearch2317<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2318: CUDA_DES_PerformSearch2318<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2319: CUDA_DES_PerformSearch2319<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2320: CUDA_DES_PerformSearch2320<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2321: CUDA_DES_PerformSearch2321<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2322: CUDA_DES_PerformSearch2322<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2323: CUDA_DES_PerformSearch2323<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2324: CUDA_DES_PerformSearch2324<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2325: CUDA_DES_PerformSearch2325<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2326: CUDA_DES_PerformSearch2326<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2327: CUDA_DES_PerformSearch2327<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2328: CUDA_DES_PerformSearch2328<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2329: CUDA_DES_PerformSearch2329<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2330: CUDA_DES_PerformSearch2330<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2331: CUDA_DES_PerformSearch2331<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2332: CUDA_DES_PerformSearch2332<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2333: CUDA_DES_PerformSearch2333<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2334: CUDA_DES_PerformSearch2334<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2335: CUDA_DES_PerformSearch2335<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2336: CUDA_DES_PerformSearch2336<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2337: CUDA_DES_PerformSearch2337<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2338: CUDA_DES_PerformSearch2338<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2339: CUDA_DES_PerformSearch2339<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2340: CUDA_DES_PerformSearch2340<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2341: CUDA_DES_PerformSearch2341<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2342: CUDA_DES_PerformSearch2342<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2343: CUDA_DES_PerformSearch2343<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2344: CUDA_DES_PerformSearch2344<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2345: CUDA_DES_PerformSearch2345<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2346: CUDA_DES_PerformSearch2346<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2347: CUDA_DES_PerformSearch2347<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2348: CUDA_DES_PerformSearch2348<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2349: CUDA_DES_PerformSearch2349<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2350: CUDA_DES_PerformSearch2350<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2351: CUDA_DES_PerformSearch2351<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2352: CUDA_DES_PerformSearch2352<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2353: CUDA_DES_PerformSearch2353<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2354: CUDA_DES_PerformSearch2354<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2355: CUDA_DES_PerformSearch2355<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2356: CUDA_DES_PerformSearch2356<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2357: CUDA_DES_PerformSearch2357<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2358: CUDA_DES_PerformSearch2358<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2359: CUDA_DES_PerformSearch2359<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2360: CUDA_DES_PerformSearch2360<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2361: CUDA_DES_PerformSearch2361<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2362: CUDA_DES_PerformSearch2362<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2363: CUDA_DES_PerformSearch2363<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2364: CUDA_DES_PerformSearch2364<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2365: CUDA_DES_PerformSearch2365<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2366: CUDA_DES_PerformSearch2366<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2367: CUDA_DES_PerformSearch2367<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2368: CUDA_DES_PerformSearch2368<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2369: CUDA_DES_PerformSearch2369<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2370: CUDA_DES_PerformSearch2370<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2371: CUDA_DES_PerformSearch2371<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2372: CUDA_DES_PerformSearch2372<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2373: CUDA_DES_PerformSearch2373<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2374: CUDA_DES_PerformSearch2374<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2375: CUDA_DES_PerformSearch2375<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2376: CUDA_DES_PerformSearch2376<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2377: CUDA_DES_PerformSearch2377<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2378: CUDA_DES_PerformSearch2378<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2379: CUDA_DES_PerformSearch2379<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2380: CUDA_DES_PerformSearch2380<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2381: CUDA_DES_PerformSearch2381<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2382: CUDA_DES_PerformSearch2382<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2383: CUDA_DES_PerformSearch2383<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2384: CUDA_DES_PerformSearch2384<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2385: CUDA_DES_PerformSearch2385<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2386: CUDA_DES_PerformSearch2386<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2387: CUDA_DES_PerformSearch2387<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2388: CUDA_DES_PerformSearch2388<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2389: CUDA_DES_PerformSearch2389<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2390: CUDA_DES_PerformSearch2390<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2391: CUDA_DES_PerformSearch2391<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2392: CUDA_DES_PerformSearch2392<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2393: CUDA_DES_PerformSearch2393<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2394: CUDA_DES_PerformSearch2394<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2395: CUDA_DES_PerformSearch2395<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2396: CUDA_DES_PerformSearch2396<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2397: CUDA_DES_PerformSearch2397<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2398: CUDA_DES_PerformSearch2398<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2399: CUDA_DES_PerformSearch2399<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2400: CUDA_DES_PerformSearch2400<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2401: CUDA_DES_PerformSearch2401<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2402: CUDA_DES_PerformSearch2402<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2403: CUDA_DES_PerformSearch2403<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2404: CUDA_DES_PerformSearch2404<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2405: CUDA_DES_PerformSearch2405<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2406: CUDA_DES_PerformSearch2406<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2407: CUDA_DES_PerformSearch2407<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2408: CUDA_DES_PerformSearch2408<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2409: CUDA_DES_PerformSearch2409<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2410: CUDA_DES_PerformSearch2410<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2411: CUDA_DES_PerformSearch2411<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2412: CUDA_DES_PerformSearch2412<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2413: CUDA_DES_PerformSearch2413<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2414: CUDA_DES_PerformSearch2414<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2415: CUDA_DES_PerformSearch2415<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2416: CUDA_DES_PerformSearch2416<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2417: CUDA_DES_PerformSearch2417<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2418: CUDA_DES_PerformSearch2418<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2419: CUDA_DES_PerformSearch2419<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2420: CUDA_DES_PerformSearch2420<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2421: CUDA_DES_PerformSearch2421<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2422: CUDA_DES_PerformSearch2422<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2423: CUDA_DES_PerformSearch2423<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2424: CUDA_DES_PerformSearch2424<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2425: CUDA_DES_PerformSearch2425<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2426: CUDA_DES_PerformSearch2426<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2427: CUDA_DES_PerformSearch2427<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2428: CUDA_DES_PerformSearch2428<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2429: CUDA_DES_PerformSearch2429<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2430: CUDA_DES_PerformSearch2430<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2431: CUDA_DES_PerformSearch2431<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2432: CUDA_DES_PerformSearch2432<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2433: CUDA_DES_PerformSearch2433<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2434: CUDA_DES_PerformSearch2434<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2435: CUDA_DES_PerformSearch2435<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2436: CUDA_DES_PerformSearch2436<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2437: CUDA_DES_PerformSearch2437<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2438: CUDA_DES_PerformSearch2438<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2439: CUDA_DES_PerformSearch2439<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2440: CUDA_DES_PerformSearch2440<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2441: CUDA_DES_PerformSearch2441<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2442: CUDA_DES_PerformSearch2442<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2443: CUDA_DES_PerformSearch2443<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2444: CUDA_DES_PerformSearch2444<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2445: CUDA_DES_PerformSearch2445<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2446: CUDA_DES_PerformSearch2446<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2447: CUDA_DES_PerformSearch2447<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2448: CUDA_DES_PerformSearch2448<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2449: CUDA_DES_PerformSearch2449<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2450: CUDA_DES_PerformSearch2450<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2451: CUDA_DES_PerformSearch2451<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2452: CUDA_DES_PerformSearch2452<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2453: CUDA_DES_PerformSearch2453<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2454: CUDA_DES_PerformSearch2454<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2455: CUDA_DES_PerformSearch2455<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2456: CUDA_DES_PerformSearch2456<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2457: CUDA_DES_PerformSearch2457<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2458: CUDA_DES_PerformSearch2458<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2459: CUDA_DES_PerformSearch2459<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2460: CUDA_DES_PerformSearch2460<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2461: CUDA_DES_PerformSearch2461<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2462: CUDA_DES_PerformSearch2462<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2463: CUDA_DES_PerformSearch2463<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2464: CUDA_DES_PerformSearch2464<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2465: CUDA_DES_PerformSearch2465<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2466: CUDA_DES_PerformSearch2466<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2467: CUDA_DES_PerformSearch2467<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2468: CUDA_DES_PerformSearch2468<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2469: CUDA_DES_PerformSearch2469<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2470: CUDA_DES_PerformSearch2470<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2471: CUDA_DES_PerformSearch2471<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2472: CUDA_DES_PerformSearch2472<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2473: CUDA_DES_PerformSearch2473<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2474: CUDA_DES_PerformSearch2474<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2475: CUDA_DES_PerformSearch2475<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2476: CUDA_DES_PerformSearch2476<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2477: CUDA_DES_PerformSearch2477<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2478: CUDA_DES_PerformSearch2478<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2479: CUDA_DES_PerformSearch2479<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2480: CUDA_DES_PerformSearch2480<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2481: CUDA_DES_PerformSearch2481<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2482: CUDA_DES_PerformSearch2482<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2483: CUDA_DES_PerformSearch2483<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2484: CUDA_DES_PerformSearch2484<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2485: CUDA_DES_PerformSearch2485<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2486: CUDA_DES_PerformSearch2486<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2487: CUDA_DES_PerformSearch2487<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2488: CUDA_DES_PerformSearch2488<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2489: CUDA_DES_PerformSearch2489<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2490: CUDA_DES_PerformSearch2490<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2491: CUDA_DES_PerformSearch2491<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2492: CUDA_DES_PerformSearch2492<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2493: CUDA_DES_PerformSearch2493<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2494: CUDA_DES_PerformSearch2494<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2495: CUDA_DES_PerformSearch2495<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2496: CUDA_DES_PerformSearch2496<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2497: CUDA_DES_PerformSearch2497<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2498: CUDA_DES_PerformSearch2498<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2499: CUDA_DES_PerformSearch2499<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2500: CUDA_DES_PerformSearch2500<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2501: CUDA_DES_PerformSearch2501<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2502: CUDA_DES_PerformSearch2502<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2503: CUDA_DES_PerformSearch2503<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2504: CUDA_DES_PerformSearch2504<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2505: CUDA_DES_PerformSearch2505<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2506: CUDA_DES_PerformSearch2506<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2507: CUDA_DES_PerformSearch2507<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2508: CUDA_DES_PerformSearch2508<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2509: CUDA_DES_PerformSearch2509<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2510: CUDA_DES_PerformSearch2510<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2511: CUDA_DES_PerformSearch2511<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2512: CUDA_DES_PerformSearch2512<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2513: CUDA_DES_PerformSearch2513<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2514: CUDA_DES_PerformSearch2514<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2515: CUDA_DES_PerformSearch2515<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2516: CUDA_DES_PerformSearch2516<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2517: CUDA_DES_PerformSearch2517<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2518: CUDA_DES_PerformSearch2518<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2519: CUDA_DES_PerformSearch2519<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2520: CUDA_DES_PerformSearch2520<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2521: CUDA_DES_PerformSearch2521<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2522: CUDA_DES_PerformSearch2522<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2523: CUDA_DES_PerformSearch2523<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2524: CUDA_DES_PerformSearch2524<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2525: CUDA_DES_PerformSearch2525<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2526: CUDA_DES_PerformSearch2526<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2527: CUDA_DES_PerformSearch2527<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2528: CUDA_DES_PerformSearch2528<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2529: CUDA_DES_PerformSearch2529<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2530: CUDA_DES_PerformSearch2530<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2531: CUDA_DES_PerformSearch2531<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2532: CUDA_DES_PerformSearch2532<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2533: CUDA_DES_PerformSearch2533<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2534: CUDA_DES_PerformSearch2534<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2535: CUDA_DES_PerformSearch2535<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2536: CUDA_DES_PerformSearch2536<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2537: CUDA_DES_PerformSearch2537<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2538: CUDA_DES_PerformSearch2538<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2539: CUDA_DES_PerformSearch2539<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2540: CUDA_DES_PerformSearch2540<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2541: CUDA_DES_PerformSearch2541<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2542: CUDA_DES_PerformSearch2542<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2543: CUDA_DES_PerformSearch2543<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2544: CUDA_DES_PerformSearch2544<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2545: CUDA_DES_PerformSearch2545<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2546: CUDA_DES_PerformSearch2546<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2547: CUDA_DES_PerformSearch2547<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2548: CUDA_DES_PerformSearch2548<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2549: CUDA_DES_PerformSearch2549<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2550: CUDA_DES_PerformSearch2550<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2551: CUDA_DES_PerformSearch2551<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2552: CUDA_DES_PerformSearch2552<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2553: CUDA_DES_PerformSearch2553<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2554: CUDA_DES_PerformSearch2554<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2555: CUDA_DES_PerformSearch2555<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2556: CUDA_DES_PerformSearch2556<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2557: CUDA_DES_PerformSearch2557<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2558: CUDA_DES_PerformSearch2558<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	case 2559: CUDA_DES_PerformSearch2559<<<dimGrid, dimBlock>>>(CUDA_DES_PERFORM_SEARCH_ARGS); break;
	}
}
