// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 1536
#include "CUDA10_Registers_Kernel.h"
#define SALT 1537
#include "CUDA10_Registers_Kernel.h"
#define SALT 1538
#include "CUDA10_Registers_Kernel.h"
#define SALT 1539
#include "CUDA10_Registers_Kernel.h"
#define SALT 1540
#include "CUDA10_Registers_Kernel.h"
#define SALT 1541
#include "CUDA10_Registers_Kernel.h"
#define SALT 1542
#include "CUDA10_Registers_Kernel.h"
#define SALT 1543
#include "CUDA10_Registers_Kernel.h"
#define SALT 1544
#include "CUDA10_Registers_Kernel.h"
#define SALT 1545
#include "CUDA10_Registers_Kernel.h"
#define SALT 1546
#include "CUDA10_Registers_Kernel.h"
#define SALT 1547
#include "CUDA10_Registers_Kernel.h"
#define SALT 1548
#include "CUDA10_Registers_Kernel.h"
#define SALT 1549
#include "CUDA10_Registers_Kernel.h"
#define SALT 1550
#include "CUDA10_Registers_Kernel.h"
#define SALT 1551
#include "CUDA10_Registers_Kernel.h"
#define SALT 1552
#include "CUDA10_Registers_Kernel.h"
#define SALT 1553
#include "CUDA10_Registers_Kernel.h"
#define SALT 1554
#include "CUDA10_Registers_Kernel.h"
#define SALT 1555
#include "CUDA10_Registers_Kernel.h"
#define SALT 1556
#include "CUDA10_Registers_Kernel.h"
#define SALT 1557
#include "CUDA10_Registers_Kernel.h"
#define SALT 1558
#include "CUDA10_Registers_Kernel.h"
#define SALT 1559
#include "CUDA10_Registers_Kernel.h"
#define SALT 1560
#include "CUDA10_Registers_Kernel.h"
#define SALT 1561
#include "CUDA10_Registers_Kernel.h"
#define SALT 1562
#include "CUDA10_Registers_Kernel.h"
#define SALT 1563
#include "CUDA10_Registers_Kernel.h"
#define SALT 1564
#include "CUDA10_Registers_Kernel.h"
#define SALT 1565
#include "CUDA10_Registers_Kernel.h"
#define SALT 1566
#include "CUDA10_Registers_Kernel.h"
#define SALT 1567
#include "CUDA10_Registers_Kernel.h"
#define SALT 1568
#include "CUDA10_Registers_Kernel.h"
#define SALT 1569
#include "CUDA10_Registers_Kernel.h"
#define SALT 1570
#include "CUDA10_Registers_Kernel.h"
#define SALT 1571
#include "CUDA10_Registers_Kernel.h"
#define SALT 1572
#include "CUDA10_Registers_Kernel.h"
#define SALT 1573
#include "CUDA10_Registers_Kernel.h"
#define SALT 1574
#include "CUDA10_Registers_Kernel.h"
#define SALT 1575
#include "CUDA10_Registers_Kernel.h"
#define SALT 1576
#include "CUDA10_Registers_Kernel.h"
#define SALT 1577
#include "CUDA10_Registers_Kernel.h"
#define SALT 1578
#include "CUDA10_Registers_Kernel.h"
#define SALT 1579
#include "CUDA10_Registers_Kernel.h"
#define SALT 1580
#include "CUDA10_Registers_Kernel.h"
#define SALT 1581
#include "CUDA10_Registers_Kernel.h"
#define SALT 1582
#include "CUDA10_Registers_Kernel.h"
#define SALT 1583
#include "CUDA10_Registers_Kernel.h"
#define SALT 1584
#include "CUDA10_Registers_Kernel.h"
#define SALT 1585
#include "CUDA10_Registers_Kernel.h"
#define SALT 1586
#include "CUDA10_Registers_Kernel.h"
#define SALT 1587
#include "CUDA10_Registers_Kernel.h"
#define SALT 1588
#include "CUDA10_Registers_Kernel.h"
#define SALT 1589
#include "CUDA10_Registers_Kernel.h"
#define SALT 1590
#include "CUDA10_Registers_Kernel.h"
#define SALT 1591
#include "CUDA10_Registers_Kernel.h"
#define SALT 1592
#include "CUDA10_Registers_Kernel.h"
#define SALT 1593
#include "CUDA10_Registers_Kernel.h"
#define SALT 1594
#include "CUDA10_Registers_Kernel.h"
#define SALT 1595
#include "CUDA10_Registers_Kernel.h"
#define SALT 1596
#include "CUDA10_Registers_Kernel.h"
#define SALT 1597
#include "CUDA10_Registers_Kernel.h"
#define SALT 1598
#include "CUDA10_Registers_Kernel.h"
#define SALT 1599
#include "CUDA10_Registers_Kernel.h"
#define SALT 1600
#include "CUDA10_Registers_Kernel.h"
#define SALT 1601
#include "CUDA10_Registers_Kernel.h"
#define SALT 1602
#include "CUDA10_Registers_Kernel.h"
#define SALT 1603
#include "CUDA10_Registers_Kernel.h"
#define SALT 1604
#include "CUDA10_Registers_Kernel.h"
#define SALT 1605
#include "CUDA10_Registers_Kernel.h"
#define SALT 1606
#include "CUDA10_Registers_Kernel.h"
#define SALT 1607
#include "CUDA10_Registers_Kernel.h"
#define SALT 1608
#include "CUDA10_Registers_Kernel.h"
#define SALT 1609
#include "CUDA10_Registers_Kernel.h"
#define SALT 1610
#include "CUDA10_Registers_Kernel.h"
#define SALT 1611
#include "CUDA10_Registers_Kernel.h"
#define SALT 1612
#include "CUDA10_Registers_Kernel.h"
#define SALT 1613
#include "CUDA10_Registers_Kernel.h"
#define SALT 1614
#include "CUDA10_Registers_Kernel.h"
#define SALT 1615
#include "CUDA10_Registers_Kernel.h"
#define SALT 1616
#include "CUDA10_Registers_Kernel.h"
#define SALT 1617
#include "CUDA10_Registers_Kernel.h"
#define SALT 1618
#include "CUDA10_Registers_Kernel.h"
#define SALT 1619
#include "CUDA10_Registers_Kernel.h"
#define SALT 1620
#include "CUDA10_Registers_Kernel.h"
#define SALT 1621
#include "CUDA10_Registers_Kernel.h"
#define SALT 1622
#include "CUDA10_Registers_Kernel.h"
#define SALT 1623
#include "CUDA10_Registers_Kernel.h"
#define SALT 1624
#include "CUDA10_Registers_Kernel.h"
#define SALT 1625
#include "CUDA10_Registers_Kernel.h"
#define SALT 1626
#include "CUDA10_Registers_Kernel.h"
#define SALT 1627
#include "CUDA10_Registers_Kernel.h"
#define SALT 1628
#include "CUDA10_Registers_Kernel.h"
#define SALT 1629
#include "CUDA10_Registers_Kernel.h"
#define SALT 1630
#include "CUDA10_Registers_Kernel.h"
#define SALT 1631
#include "CUDA10_Registers_Kernel.h"
#define SALT 1632
#include "CUDA10_Registers_Kernel.h"
#define SALT 1633
#include "CUDA10_Registers_Kernel.h"
#define SALT 1634
#include "CUDA10_Registers_Kernel.h"
#define SALT 1635
#include "CUDA10_Registers_Kernel.h"
#define SALT 1636
#include "CUDA10_Registers_Kernel.h"
#define SALT 1637
#include "CUDA10_Registers_Kernel.h"
#define SALT 1638
#include "CUDA10_Registers_Kernel.h"
#define SALT 1639
#include "CUDA10_Registers_Kernel.h"
#define SALT 1640
#include "CUDA10_Registers_Kernel.h"
#define SALT 1641
#include "CUDA10_Registers_Kernel.h"
#define SALT 1642
#include "CUDA10_Registers_Kernel.h"
#define SALT 1643
#include "CUDA10_Registers_Kernel.h"
#define SALT 1644
#include "CUDA10_Registers_Kernel.h"
#define SALT 1645
#include "CUDA10_Registers_Kernel.h"
#define SALT 1646
#include "CUDA10_Registers_Kernel.h"
#define SALT 1647
#include "CUDA10_Registers_Kernel.h"
#define SALT 1648
#include "CUDA10_Registers_Kernel.h"
#define SALT 1649
#include "CUDA10_Registers_Kernel.h"
#define SALT 1650
#include "CUDA10_Registers_Kernel.h"
#define SALT 1651
#include "CUDA10_Registers_Kernel.h"
#define SALT 1652
#include "CUDA10_Registers_Kernel.h"
#define SALT 1653
#include "CUDA10_Registers_Kernel.h"
#define SALT 1654
#include "CUDA10_Registers_Kernel.h"
#define SALT 1655
#include "CUDA10_Registers_Kernel.h"
#define SALT 1656
#include "CUDA10_Registers_Kernel.h"
#define SALT 1657
#include "CUDA10_Registers_Kernel.h"
#define SALT 1658
#include "CUDA10_Registers_Kernel.h"
#define SALT 1659
#include "CUDA10_Registers_Kernel.h"
#define SALT 1660
#include "CUDA10_Registers_Kernel.h"
#define SALT 1661
#include "CUDA10_Registers_Kernel.h"
#define SALT 1662
#include "CUDA10_Registers_Kernel.h"
#define SALT 1663
#include "CUDA10_Registers_Kernel.h"
#define SALT 1664
#include "CUDA10_Registers_Kernel.h"
#define SALT 1665
#include "CUDA10_Registers_Kernel.h"
#define SALT 1666
#include "CUDA10_Registers_Kernel.h"
#define SALT 1667
#include "CUDA10_Registers_Kernel.h"
#define SALT 1668
#include "CUDA10_Registers_Kernel.h"
#define SALT 1669
#include "CUDA10_Registers_Kernel.h"
#define SALT 1670
#include "CUDA10_Registers_Kernel.h"
#define SALT 1671
#include "CUDA10_Registers_Kernel.h"
#define SALT 1672
#include "CUDA10_Registers_Kernel.h"
#define SALT 1673
#include "CUDA10_Registers_Kernel.h"
#define SALT 1674
#include "CUDA10_Registers_Kernel.h"
#define SALT 1675
#include "CUDA10_Registers_Kernel.h"
#define SALT 1676
#include "CUDA10_Registers_Kernel.h"
#define SALT 1677
#include "CUDA10_Registers_Kernel.h"
#define SALT 1678
#include "CUDA10_Registers_Kernel.h"
#define SALT 1679
#include "CUDA10_Registers_Kernel.h"
#define SALT 1680
#include "CUDA10_Registers_Kernel.h"
#define SALT 1681
#include "CUDA10_Registers_Kernel.h"
#define SALT 1682
#include "CUDA10_Registers_Kernel.h"
#define SALT 1683
#include "CUDA10_Registers_Kernel.h"
#define SALT 1684
#include "CUDA10_Registers_Kernel.h"
#define SALT 1685
#include "CUDA10_Registers_Kernel.h"
#define SALT 1686
#include "CUDA10_Registers_Kernel.h"
#define SALT 1687
#include "CUDA10_Registers_Kernel.h"
#define SALT 1688
#include "CUDA10_Registers_Kernel.h"
#define SALT 1689
#include "CUDA10_Registers_Kernel.h"
#define SALT 1690
#include "CUDA10_Registers_Kernel.h"
#define SALT 1691
#include "CUDA10_Registers_Kernel.h"
#define SALT 1692
#include "CUDA10_Registers_Kernel.h"
#define SALT 1693
#include "CUDA10_Registers_Kernel.h"
#define SALT 1694
#include "CUDA10_Registers_Kernel.h"
#define SALT 1695
#include "CUDA10_Registers_Kernel.h"
#define SALT 1696
#include "CUDA10_Registers_Kernel.h"
#define SALT 1697
#include "CUDA10_Registers_Kernel.h"
#define SALT 1698
#include "CUDA10_Registers_Kernel.h"
#define SALT 1699
#include "CUDA10_Registers_Kernel.h"
#define SALT 1700
#include "CUDA10_Registers_Kernel.h"
#define SALT 1701
#include "CUDA10_Registers_Kernel.h"
#define SALT 1702
#include "CUDA10_Registers_Kernel.h"
#define SALT 1703
#include "CUDA10_Registers_Kernel.h"
#define SALT 1704
#include "CUDA10_Registers_Kernel.h"
#define SALT 1705
#include "CUDA10_Registers_Kernel.h"
#define SALT 1706
#include "CUDA10_Registers_Kernel.h"
#define SALT 1707
#include "CUDA10_Registers_Kernel.h"
#define SALT 1708
#include "CUDA10_Registers_Kernel.h"
#define SALT 1709
#include "CUDA10_Registers_Kernel.h"
#define SALT 1710
#include "CUDA10_Registers_Kernel.h"
#define SALT 1711
#include "CUDA10_Registers_Kernel.h"
#define SALT 1712
#include "CUDA10_Registers_Kernel.h"
#define SALT 1713
#include "CUDA10_Registers_Kernel.h"
#define SALT 1714
#include "CUDA10_Registers_Kernel.h"
#define SALT 1715
#include "CUDA10_Registers_Kernel.h"
#define SALT 1716
#include "CUDA10_Registers_Kernel.h"
#define SALT 1717
#include "CUDA10_Registers_Kernel.h"
#define SALT 1718
#include "CUDA10_Registers_Kernel.h"
#define SALT 1719
#include "CUDA10_Registers_Kernel.h"
#define SALT 1720
#include "CUDA10_Registers_Kernel.h"
#define SALT 1721
#include "CUDA10_Registers_Kernel.h"
#define SALT 1722
#include "CUDA10_Registers_Kernel.h"
#define SALT 1723
#include "CUDA10_Registers_Kernel.h"
#define SALT 1724
#include "CUDA10_Registers_Kernel.h"
#define SALT 1725
#include "CUDA10_Registers_Kernel.h"
#define SALT 1726
#include "CUDA10_Registers_Kernel.h"
#define SALT 1727
#include "CUDA10_Registers_Kernel.h"
#define SALT 1728
#include "CUDA10_Registers_Kernel.h"
#define SALT 1729
#include "CUDA10_Registers_Kernel.h"
#define SALT 1730
#include "CUDA10_Registers_Kernel.h"
#define SALT 1731
#include "CUDA10_Registers_Kernel.h"
#define SALT 1732
#include "CUDA10_Registers_Kernel.h"
#define SALT 1733
#include "CUDA10_Registers_Kernel.h"
#define SALT 1734
#include "CUDA10_Registers_Kernel.h"
#define SALT 1735
#include "CUDA10_Registers_Kernel.h"
#define SALT 1736
#include "CUDA10_Registers_Kernel.h"
#define SALT 1737
#include "CUDA10_Registers_Kernel.h"
#define SALT 1738
#include "CUDA10_Registers_Kernel.h"
#define SALT 1739
#include "CUDA10_Registers_Kernel.h"
#define SALT 1740
#include "CUDA10_Registers_Kernel.h"
#define SALT 1741
#include "CUDA10_Registers_Kernel.h"
#define SALT 1742
#include "CUDA10_Registers_Kernel.h"
#define SALT 1743
#include "CUDA10_Registers_Kernel.h"
#define SALT 1744
#include "CUDA10_Registers_Kernel.h"
#define SALT 1745
#include "CUDA10_Registers_Kernel.h"
#define SALT 1746
#include "CUDA10_Registers_Kernel.h"
#define SALT 1747
#include "CUDA10_Registers_Kernel.h"
#define SALT 1748
#include "CUDA10_Registers_Kernel.h"
#define SALT 1749
#include "CUDA10_Registers_Kernel.h"
#define SALT 1750
#include "CUDA10_Registers_Kernel.h"
#define SALT 1751
#include "CUDA10_Registers_Kernel.h"
#define SALT 1752
#include "CUDA10_Registers_Kernel.h"
#define SALT 1753
#include "CUDA10_Registers_Kernel.h"
#define SALT 1754
#include "CUDA10_Registers_Kernel.h"
#define SALT 1755
#include "CUDA10_Registers_Kernel.h"
#define SALT 1756
#include "CUDA10_Registers_Kernel.h"
#define SALT 1757
#include "CUDA10_Registers_Kernel.h"
#define SALT 1758
#include "CUDA10_Registers_Kernel.h"
#define SALT 1759
#include "CUDA10_Registers_Kernel.h"
#define SALT 1760
#include "CUDA10_Registers_Kernel.h"
#define SALT 1761
#include "CUDA10_Registers_Kernel.h"
#define SALT 1762
#include "CUDA10_Registers_Kernel.h"
#define SALT 1763
#include "CUDA10_Registers_Kernel.h"
#define SALT 1764
#include "CUDA10_Registers_Kernel.h"
#define SALT 1765
#include "CUDA10_Registers_Kernel.h"
#define SALT 1766
#include "CUDA10_Registers_Kernel.h"
#define SALT 1767
#include "CUDA10_Registers_Kernel.h"
#define SALT 1768
#include "CUDA10_Registers_Kernel.h"
#define SALT 1769
#include "CUDA10_Registers_Kernel.h"
#define SALT 1770
#include "CUDA10_Registers_Kernel.h"
#define SALT 1771
#include "CUDA10_Registers_Kernel.h"
#define SALT 1772
#include "CUDA10_Registers_Kernel.h"
#define SALT 1773
#include "CUDA10_Registers_Kernel.h"
#define SALT 1774
#include "CUDA10_Registers_Kernel.h"
#define SALT 1775
#include "CUDA10_Registers_Kernel.h"
#define SALT 1776
#include "CUDA10_Registers_Kernel.h"
#define SALT 1777
#include "CUDA10_Registers_Kernel.h"
#define SALT 1778
#include "CUDA10_Registers_Kernel.h"
#define SALT 1779
#include "CUDA10_Registers_Kernel.h"
#define SALT 1780
#include "CUDA10_Registers_Kernel.h"
#define SALT 1781
#include "CUDA10_Registers_Kernel.h"
#define SALT 1782
#include "CUDA10_Registers_Kernel.h"
#define SALT 1783
#include "CUDA10_Registers_Kernel.h"
#define SALT 1784
#include "CUDA10_Registers_Kernel.h"
#define SALT 1785
#include "CUDA10_Registers_Kernel.h"
#define SALT 1786
#include "CUDA10_Registers_Kernel.h"
#define SALT 1787
#include "CUDA10_Registers_Kernel.h"
#define SALT 1788
#include "CUDA10_Registers_Kernel.h"
#define SALT 1789
#include "CUDA10_Registers_Kernel.h"
#define SALT 1790
#include "CUDA10_Registers_Kernel.h"
#define SALT 1791
#include "CUDA10_Registers_Kernel.h"
#define SALT 1792
#include "CUDA10_Registers_Kernel.h"
#define SALT 1793
#include "CUDA10_Registers_Kernel.h"
#define SALT 1794
#include "CUDA10_Registers_Kernel.h"
#define SALT 1795
#include "CUDA10_Registers_Kernel.h"
#define SALT 1796
#include "CUDA10_Registers_Kernel.h"
#define SALT 1797
#include "CUDA10_Registers_Kernel.h"
#define SALT 1798
#include "CUDA10_Registers_Kernel.h"
#define SALT 1799
#include "CUDA10_Registers_Kernel.h"
#define SALT 1800
#include "CUDA10_Registers_Kernel.h"
#define SALT 1801
#include "CUDA10_Registers_Kernel.h"
#define SALT 1802
#include "CUDA10_Registers_Kernel.h"
#define SALT 1803
#include "CUDA10_Registers_Kernel.h"
#define SALT 1804
#include "CUDA10_Registers_Kernel.h"
#define SALT 1805
#include "CUDA10_Registers_Kernel.h"
#define SALT 1806
#include "CUDA10_Registers_Kernel.h"
#define SALT 1807
#include "CUDA10_Registers_Kernel.h"
#define SALT 1808
#include "CUDA10_Registers_Kernel.h"
#define SALT 1809
#include "CUDA10_Registers_Kernel.h"
#define SALT 1810
#include "CUDA10_Registers_Kernel.h"
#define SALT 1811
#include "CUDA10_Registers_Kernel.h"
#define SALT 1812
#include "CUDA10_Registers_Kernel.h"
#define SALT 1813
#include "CUDA10_Registers_Kernel.h"
#define SALT 1814
#include "CUDA10_Registers_Kernel.h"
#define SALT 1815
#include "CUDA10_Registers_Kernel.h"
#define SALT 1816
#include "CUDA10_Registers_Kernel.h"
#define SALT 1817
#include "CUDA10_Registers_Kernel.h"
#define SALT 1818
#include "CUDA10_Registers_Kernel.h"
#define SALT 1819
#include "CUDA10_Registers_Kernel.h"
#define SALT 1820
#include "CUDA10_Registers_Kernel.h"
#define SALT 1821
#include "CUDA10_Registers_Kernel.h"
#define SALT 1822
#include "CUDA10_Registers_Kernel.h"
#define SALT 1823
#include "CUDA10_Registers_Kernel.h"
#define SALT 1824
#include "CUDA10_Registers_Kernel.h"
#define SALT 1825
#include "CUDA10_Registers_Kernel.h"
#define SALT 1826
#include "CUDA10_Registers_Kernel.h"
#define SALT 1827
#include "CUDA10_Registers_Kernel.h"
#define SALT 1828
#include "CUDA10_Registers_Kernel.h"
#define SALT 1829
#include "CUDA10_Registers_Kernel.h"
#define SALT 1830
#include "CUDA10_Registers_Kernel.h"
#define SALT 1831
#include "CUDA10_Registers_Kernel.h"
#define SALT 1832
#include "CUDA10_Registers_Kernel.h"
#define SALT 1833
#include "CUDA10_Registers_Kernel.h"
#define SALT 1834
#include "CUDA10_Registers_Kernel.h"
#define SALT 1835
#include "CUDA10_Registers_Kernel.h"
#define SALT 1836
#include "CUDA10_Registers_Kernel.h"
#define SALT 1837
#include "CUDA10_Registers_Kernel.h"
#define SALT 1838
#include "CUDA10_Registers_Kernel.h"
#define SALT 1839
#include "CUDA10_Registers_Kernel.h"
#define SALT 1840
#include "CUDA10_Registers_Kernel.h"
#define SALT 1841
#include "CUDA10_Registers_Kernel.h"
#define SALT 1842
#include "CUDA10_Registers_Kernel.h"
#define SALT 1843
#include "CUDA10_Registers_Kernel.h"
#define SALT 1844
#include "CUDA10_Registers_Kernel.h"
#define SALT 1845
#include "CUDA10_Registers_Kernel.h"
#define SALT 1846
#include "CUDA10_Registers_Kernel.h"
#define SALT 1847
#include "CUDA10_Registers_Kernel.h"
#define SALT 1848
#include "CUDA10_Registers_Kernel.h"
#define SALT 1849
#include "CUDA10_Registers_Kernel.h"
#define SALT 1850
#include "CUDA10_Registers_Kernel.h"
#define SALT 1851
#include "CUDA10_Registers_Kernel.h"
#define SALT 1852
#include "CUDA10_Registers_Kernel.h"
#define SALT 1853
#include "CUDA10_Registers_Kernel.h"
#define SALT 1854
#include "CUDA10_Registers_Kernel.h"
#define SALT 1855
#include "CUDA10_Registers_Kernel.h"
#define SALT 1856
#include "CUDA10_Registers_Kernel.h"
#define SALT 1857
#include "CUDA10_Registers_Kernel.h"
#define SALT 1858
#include "CUDA10_Registers_Kernel.h"
#define SALT 1859
#include "CUDA10_Registers_Kernel.h"
#define SALT 1860
#include "CUDA10_Registers_Kernel.h"
#define SALT 1861
#include "CUDA10_Registers_Kernel.h"
#define SALT 1862
#include "CUDA10_Registers_Kernel.h"
#define SALT 1863
#include "CUDA10_Registers_Kernel.h"
#define SALT 1864
#include "CUDA10_Registers_Kernel.h"
#define SALT 1865
#include "CUDA10_Registers_Kernel.h"
#define SALT 1866
#include "CUDA10_Registers_Kernel.h"
#define SALT 1867
#include "CUDA10_Registers_Kernel.h"
#define SALT 1868
#include "CUDA10_Registers_Kernel.h"
#define SALT 1869
#include "CUDA10_Registers_Kernel.h"
#define SALT 1870
#include "CUDA10_Registers_Kernel.h"
#define SALT 1871
#include "CUDA10_Registers_Kernel.h"
#define SALT 1872
#include "CUDA10_Registers_Kernel.h"
#define SALT 1873
#include "CUDA10_Registers_Kernel.h"
#define SALT 1874
#include "CUDA10_Registers_Kernel.h"
#define SALT 1875
#include "CUDA10_Registers_Kernel.h"
#define SALT 1876
#include "CUDA10_Registers_Kernel.h"
#define SALT 1877
#include "CUDA10_Registers_Kernel.h"
#define SALT 1878
#include "CUDA10_Registers_Kernel.h"
#define SALT 1879
#include "CUDA10_Registers_Kernel.h"
#define SALT 1880
#include "CUDA10_Registers_Kernel.h"
#define SALT 1881
#include "CUDA10_Registers_Kernel.h"
#define SALT 1882
#include "CUDA10_Registers_Kernel.h"
#define SALT 1883
#include "CUDA10_Registers_Kernel.h"
#define SALT 1884
#include "CUDA10_Registers_Kernel.h"
#define SALT 1885
#include "CUDA10_Registers_Kernel.h"
#define SALT 1886
#include "CUDA10_Registers_Kernel.h"
#define SALT 1887
#include "CUDA10_Registers_Kernel.h"
#define SALT 1888
#include "CUDA10_Registers_Kernel.h"
#define SALT 1889
#include "CUDA10_Registers_Kernel.h"
#define SALT 1890
#include "CUDA10_Registers_Kernel.h"
#define SALT 1891
#include "CUDA10_Registers_Kernel.h"
#define SALT 1892
#include "CUDA10_Registers_Kernel.h"
#define SALT 1893
#include "CUDA10_Registers_Kernel.h"
#define SALT 1894
#include "CUDA10_Registers_Kernel.h"
#define SALT 1895
#include "CUDA10_Registers_Kernel.h"
#define SALT 1896
#include "CUDA10_Registers_Kernel.h"
#define SALT 1897
#include "CUDA10_Registers_Kernel.h"
#define SALT 1898
#include "CUDA10_Registers_Kernel.h"
#define SALT 1899
#include "CUDA10_Registers_Kernel.h"
#define SALT 1900
#include "CUDA10_Registers_Kernel.h"
#define SALT 1901
#include "CUDA10_Registers_Kernel.h"
#define SALT 1902
#include "CUDA10_Registers_Kernel.h"
#define SALT 1903
#include "CUDA10_Registers_Kernel.h"
#define SALT 1904
#include "CUDA10_Registers_Kernel.h"
#define SALT 1905
#include "CUDA10_Registers_Kernel.h"
#define SALT 1906
#include "CUDA10_Registers_Kernel.h"
#define SALT 1907
#include "CUDA10_Registers_Kernel.h"
#define SALT 1908
#include "CUDA10_Registers_Kernel.h"
#define SALT 1909
#include "CUDA10_Registers_Kernel.h"
#define SALT 1910
#include "CUDA10_Registers_Kernel.h"
#define SALT 1911
#include "CUDA10_Registers_Kernel.h"
#define SALT 1912
#include "CUDA10_Registers_Kernel.h"
#define SALT 1913
#include "CUDA10_Registers_Kernel.h"
#define SALT 1914
#include "CUDA10_Registers_Kernel.h"
#define SALT 1915
#include "CUDA10_Registers_Kernel.h"
#define SALT 1916
#include "CUDA10_Registers_Kernel.h"
#define SALT 1917
#include "CUDA10_Registers_Kernel.h"
#define SALT 1918
#include "CUDA10_Registers_Kernel.h"
#define SALT 1919
#include "CUDA10_Registers_Kernel.h"
#define SALT 1920
#include "CUDA10_Registers_Kernel.h"
#define SALT 1921
#include "CUDA10_Registers_Kernel.h"
#define SALT 1922
#include "CUDA10_Registers_Kernel.h"
#define SALT 1923
#include "CUDA10_Registers_Kernel.h"
#define SALT 1924
#include "CUDA10_Registers_Kernel.h"
#define SALT 1925
#include "CUDA10_Registers_Kernel.h"
#define SALT 1926
#include "CUDA10_Registers_Kernel.h"
#define SALT 1927
#include "CUDA10_Registers_Kernel.h"
#define SALT 1928
#include "CUDA10_Registers_Kernel.h"
#define SALT 1929
#include "CUDA10_Registers_Kernel.h"
#define SALT 1930
#include "CUDA10_Registers_Kernel.h"
#define SALT 1931
#include "CUDA10_Registers_Kernel.h"
#define SALT 1932
#include "CUDA10_Registers_Kernel.h"
#define SALT 1933
#include "CUDA10_Registers_Kernel.h"
#define SALT 1934
#include "CUDA10_Registers_Kernel.h"
#define SALT 1935
#include "CUDA10_Registers_Kernel.h"
#define SALT 1936
#include "CUDA10_Registers_Kernel.h"
#define SALT 1937
#include "CUDA10_Registers_Kernel.h"
#define SALT 1938
#include "CUDA10_Registers_Kernel.h"
#define SALT 1939
#include "CUDA10_Registers_Kernel.h"
#define SALT 1940
#include "CUDA10_Registers_Kernel.h"
#define SALT 1941
#include "CUDA10_Registers_Kernel.h"
#define SALT 1942
#include "CUDA10_Registers_Kernel.h"
#define SALT 1943
#include "CUDA10_Registers_Kernel.h"
#define SALT 1944
#include "CUDA10_Registers_Kernel.h"
#define SALT 1945
#include "CUDA10_Registers_Kernel.h"
#define SALT 1946
#include "CUDA10_Registers_Kernel.h"
#define SALT 1947
#include "CUDA10_Registers_Kernel.h"
#define SALT 1948
#include "CUDA10_Registers_Kernel.h"
#define SALT 1949
#include "CUDA10_Registers_Kernel.h"
#define SALT 1950
#include "CUDA10_Registers_Kernel.h"
#define SALT 1951
#include "CUDA10_Registers_Kernel.h"
#define SALT 1952
#include "CUDA10_Registers_Kernel.h"
#define SALT 1953
#include "CUDA10_Registers_Kernel.h"
#define SALT 1954
#include "CUDA10_Registers_Kernel.h"
#define SALT 1955
#include "CUDA10_Registers_Kernel.h"
#define SALT 1956
#include "CUDA10_Registers_Kernel.h"
#define SALT 1957
#include "CUDA10_Registers_Kernel.h"
#define SALT 1958
#include "CUDA10_Registers_Kernel.h"
#define SALT 1959
#include "CUDA10_Registers_Kernel.h"
#define SALT 1960
#include "CUDA10_Registers_Kernel.h"
#define SALT 1961
#include "CUDA10_Registers_Kernel.h"
#define SALT 1962
#include "CUDA10_Registers_Kernel.h"
#define SALT 1963
#include "CUDA10_Registers_Kernel.h"
#define SALT 1964
#include "CUDA10_Registers_Kernel.h"
#define SALT 1965
#include "CUDA10_Registers_Kernel.h"
#define SALT 1966
#include "CUDA10_Registers_Kernel.h"
#define SALT 1967
#include "CUDA10_Registers_Kernel.h"
#define SALT 1968
#include "CUDA10_Registers_Kernel.h"
#define SALT 1969
#include "CUDA10_Registers_Kernel.h"
#define SALT 1970
#include "CUDA10_Registers_Kernel.h"
#define SALT 1971
#include "CUDA10_Registers_Kernel.h"
#define SALT 1972
#include "CUDA10_Registers_Kernel.h"
#define SALT 1973
#include "CUDA10_Registers_Kernel.h"
#define SALT 1974
#include "CUDA10_Registers_Kernel.h"
#define SALT 1975
#include "CUDA10_Registers_Kernel.h"
#define SALT 1976
#include "CUDA10_Registers_Kernel.h"
#define SALT 1977
#include "CUDA10_Registers_Kernel.h"
#define SALT 1978
#include "CUDA10_Registers_Kernel.h"
#define SALT 1979
#include "CUDA10_Registers_Kernel.h"
#define SALT 1980
#include "CUDA10_Registers_Kernel.h"
#define SALT 1981
#include "CUDA10_Registers_Kernel.h"
#define SALT 1982
#include "CUDA10_Registers_Kernel.h"
#define SALT 1983
#include "CUDA10_Registers_Kernel.h"
#define SALT 1984
#include "CUDA10_Registers_Kernel.h"
#define SALT 1985
#include "CUDA10_Registers_Kernel.h"
#define SALT 1986
#include "CUDA10_Registers_Kernel.h"
#define SALT 1987
#include "CUDA10_Registers_Kernel.h"
#define SALT 1988
#include "CUDA10_Registers_Kernel.h"
#define SALT 1989
#include "CUDA10_Registers_Kernel.h"
#define SALT 1990
#include "CUDA10_Registers_Kernel.h"
#define SALT 1991
#include "CUDA10_Registers_Kernel.h"
#define SALT 1992
#include "CUDA10_Registers_Kernel.h"
#define SALT 1993
#include "CUDA10_Registers_Kernel.h"
#define SALT 1994
#include "CUDA10_Registers_Kernel.h"
#define SALT 1995
#include "CUDA10_Registers_Kernel.h"
#define SALT 1996
#include "CUDA10_Registers_Kernel.h"
#define SALT 1997
#include "CUDA10_Registers_Kernel.h"
#define SALT 1998
#include "CUDA10_Registers_Kernel.h"
#define SALT 1999
#include "CUDA10_Registers_Kernel.h"
#define SALT 2000
#include "CUDA10_Registers_Kernel.h"
#define SALT 2001
#include "CUDA10_Registers_Kernel.h"
#define SALT 2002
#include "CUDA10_Registers_Kernel.h"
#define SALT 2003
#include "CUDA10_Registers_Kernel.h"
#define SALT 2004
#include "CUDA10_Registers_Kernel.h"
#define SALT 2005
#include "CUDA10_Registers_Kernel.h"
#define SALT 2006
#include "CUDA10_Registers_Kernel.h"
#define SALT 2007
#include "CUDA10_Registers_Kernel.h"
#define SALT 2008
#include "CUDA10_Registers_Kernel.h"
#define SALT 2009
#include "CUDA10_Registers_Kernel.h"
#define SALT 2010
#include "CUDA10_Registers_Kernel.h"
#define SALT 2011
#include "CUDA10_Registers_Kernel.h"
#define SALT 2012
#include "CUDA10_Registers_Kernel.h"
#define SALT 2013
#include "CUDA10_Registers_Kernel.h"
#define SALT 2014
#include "CUDA10_Registers_Kernel.h"
#define SALT 2015
#include "CUDA10_Registers_Kernel.h"
#define SALT 2016
#include "CUDA10_Registers_Kernel.h"
#define SALT 2017
#include "CUDA10_Registers_Kernel.h"
#define SALT 2018
#include "CUDA10_Registers_Kernel.h"
#define SALT 2019
#include "CUDA10_Registers_Kernel.h"
#define SALT 2020
#include "CUDA10_Registers_Kernel.h"
#define SALT 2021
#include "CUDA10_Registers_Kernel.h"
#define SALT 2022
#include "CUDA10_Registers_Kernel.h"
#define SALT 2023
#include "CUDA10_Registers_Kernel.h"
#define SALT 2024
#include "CUDA10_Registers_Kernel.h"
#define SALT 2025
#include "CUDA10_Registers_Kernel.h"
#define SALT 2026
#include "CUDA10_Registers_Kernel.h"
#define SALT 2027
#include "CUDA10_Registers_Kernel.h"
#define SALT 2028
#include "CUDA10_Registers_Kernel.h"
#define SALT 2029
#include "CUDA10_Registers_Kernel.h"
#define SALT 2030
#include "CUDA10_Registers_Kernel.h"
#define SALT 2031
#include "CUDA10_Registers_Kernel.h"
#define SALT 2032
#include "CUDA10_Registers_Kernel.h"
#define SALT 2033
#include "CUDA10_Registers_Kernel.h"
#define SALT 2034
#include "CUDA10_Registers_Kernel.h"
#define SALT 2035
#include "CUDA10_Registers_Kernel.h"
#define SALT 2036
#include "CUDA10_Registers_Kernel.h"
#define SALT 2037
#include "CUDA10_Registers_Kernel.h"
#define SALT 2038
#include "CUDA10_Registers_Kernel.h"
#define SALT 2039
#include "CUDA10_Registers_Kernel.h"
#define SALT 2040
#include "CUDA10_Registers_Kernel.h"
#define SALT 2041
#include "CUDA10_Registers_Kernel.h"
#define SALT 2042
#include "CUDA10_Registers_Kernel.h"
#define SALT 2043
#include "CUDA10_Registers_Kernel.h"
#define SALT 2044
#include "CUDA10_Registers_Kernel.h"
#define SALT 2045
#include "CUDA10_Registers_Kernel.h"
#define SALT 2046
#include "CUDA10_Registers_Kernel.h"
#define SALT 2047
#include "CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher3()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel3(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 1536: LAUNCH_KERNEL(1536); break;
	case 1537: LAUNCH_KERNEL(1537); break;
	case 1538: LAUNCH_KERNEL(1538); break;
	case 1539: LAUNCH_KERNEL(1539); break;
	case 1540: LAUNCH_KERNEL(1540); break;
	case 1541: LAUNCH_KERNEL(1541); break;
	case 1542: LAUNCH_KERNEL(1542); break;
	case 1543: LAUNCH_KERNEL(1543); break;
	case 1544: LAUNCH_KERNEL(1544); break;
	case 1545: LAUNCH_KERNEL(1545); break;
	case 1546: LAUNCH_KERNEL(1546); break;
	case 1547: LAUNCH_KERNEL(1547); break;
	case 1548: LAUNCH_KERNEL(1548); break;
	case 1549: LAUNCH_KERNEL(1549); break;
	case 1550: LAUNCH_KERNEL(1550); break;
	case 1551: LAUNCH_KERNEL(1551); break;
	case 1552: LAUNCH_KERNEL(1552); break;
	case 1553: LAUNCH_KERNEL(1553); break;
	case 1554: LAUNCH_KERNEL(1554); break;
	case 1555: LAUNCH_KERNEL(1555); break;
	case 1556: LAUNCH_KERNEL(1556); break;
	case 1557: LAUNCH_KERNEL(1557); break;
	case 1558: LAUNCH_KERNEL(1558); break;
	case 1559: LAUNCH_KERNEL(1559); break;
	case 1560: LAUNCH_KERNEL(1560); break;
	case 1561: LAUNCH_KERNEL(1561); break;
	case 1562: LAUNCH_KERNEL(1562); break;
	case 1563: LAUNCH_KERNEL(1563); break;
	case 1564: LAUNCH_KERNEL(1564); break;
	case 1565: LAUNCH_KERNEL(1565); break;
	case 1566: LAUNCH_KERNEL(1566); break;
	case 1567: LAUNCH_KERNEL(1567); break;
	case 1568: LAUNCH_KERNEL(1568); break;
	case 1569: LAUNCH_KERNEL(1569); break;
	case 1570: LAUNCH_KERNEL(1570); break;
	case 1571: LAUNCH_KERNEL(1571); break;
	case 1572: LAUNCH_KERNEL(1572); break;
	case 1573: LAUNCH_KERNEL(1573); break;
	case 1574: LAUNCH_KERNEL(1574); break;
	case 1575: LAUNCH_KERNEL(1575); break;
	case 1576: LAUNCH_KERNEL(1576); break;
	case 1577: LAUNCH_KERNEL(1577); break;
	case 1578: LAUNCH_KERNEL(1578); break;
	case 1579: LAUNCH_KERNEL(1579); break;
	case 1580: LAUNCH_KERNEL(1580); break;
	case 1581: LAUNCH_KERNEL(1581); break;
	case 1582: LAUNCH_KERNEL(1582); break;
	case 1583: LAUNCH_KERNEL(1583); break;
	case 1584: LAUNCH_KERNEL(1584); break;
	case 1585: LAUNCH_KERNEL(1585); break;
	case 1586: LAUNCH_KERNEL(1586); break;
	case 1587: LAUNCH_KERNEL(1587); break;
	case 1588: LAUNCH_KERNEL(1588); break;
	case 1589: LAUNCH_KERNEL(1589); break;
	case 1590: LAUNCH_KERNEL(1590); break;
	case 1591: LAUNCH_KERNEL(1591); break;
	case 1592: LAUNCH_KERNEL(1592); break;
	case 1593: LAUNCH_KERNEL(1593); break;
	case 1594: LAUNCH_KERNEL(1594); break;
	case 1595: LAUNCH_KERNEL(1595); break;
	case 1596: LAUNCH_KERNEL(1596); break;
	case 1597: LAUNCH_KERNEL(1597); break;
	case 1598: LAUNCH_KERNEL(1598); break;
	case 1599: LAUNCH_KERNEL(1599); break;
	case 1600: LAUNCH_KERNEL(1600); break;
	case 1601: LAUNCH_KERNEL(1601); break;
	case 1602: LAUNCH_KERNEL(1602); break;
	case 1603: LAUNCH_KERNEL(1603); break;
	case 1604: LAUNCH_KERNEL(1604); break;
	case 1605: LAUNCH_KERNEL(1605); break;
	case 1606: LAUNCH_KERNEL(1606); break;
	case 1607: LAUNCH_KERNEL(1607); break;
	case 1608: LAUNCH_KERNEL(1608); break;
	case 1609: LAUNCH_KERNEL(1609); break;
	case 1610: LAUNCH_KERNEL(1610); break;
	case 1611: LAUNCH_KERNEL(1611); break;
	case 1612: LAUNCH_KERNEL(1612); break;
	case 1613: LAUNCH_KERNEL(1613); break;
	case 1614: LAUNCH_KERNEL(1614); break;
	case 1615: LAUNCH_KERNEL(1615); break;
	case 1616: LAUNCH_KERNEL(1616); break;
	case 1617: LAUNCH_KERNEL(1617); break;
	case 1618: LAUNCH_KERNEL(1618); break;
	case 1619: LAUNCH_KERNEL(1619); break;
	case 1620: LAUNCH_KERNEL(1620); break;
	case 1621: LAUNCH_KERNEL(1621); break;
	case 1622: LAUNCH_KERNEL(1622); break;
	case 1623: LAUNCH_KERNEL(1623); break;
	case 1624: LAUNCH_KERNEL(1624); break;
	case 1625: LAUNCH_KERNEL(1625); break;
	case 1626: LAUNCH_KERNEL(1626); break;
	case 1627: LAUNCH_KERNEL(1627); break;
	case 1628: LAUNCH_KERNEL(1628); break;
	case 1629: LAUNCH_KERNEL(1629); break;
	case 1630: LAUNCH_KERNEL(1630); break;
	case 1631: LAUNCH_KERNEL(1631); break;
	case 1632: LAUNCH_KERNEL(1632); break;
	case 1633: LAUNCH_KERNEL(1633); break;
	case 1634: LAUNCH_KERNEL(1634); break;
	case 1635: LAUNCH_KERNEL(1635); break;
	case 1636: LAUNCH_KERNEL(1636); break;
	case 1637: LAUNCH_KERNEL(1637); break;
	case 1638: LAUNCH_KERNEL(1638); break;
	case 1639: LAUNCH_KERNEL(1639); break;
	case 1640: LAUNCH_KERNEL(1640); break;
	case 1641: LAUNCH_KERNEL(1641); break;
	case 1642: LAUNCH_KERNEL(1642); break;
	case 1643: LAUNCH_KERNEL(1643); break;
	case 1644: LAUNCH_KERNEL(1644); break;
	case 1645: LAUNCH_KERNEL(1645); break;
	case 1646: LAUNCH_KERNEL(1646); break;
	case 1647: LAUNCH_KERNEL(1647); break;
	case 1648: LAUNCH_KERNEL(1648); break;
	case 1649: LAUNCH_KERNEL(1649); break;
	case 1650: LAUNCH_KERNEL(1650); break;
	case 1651: LAUNCH_KERNEL(1651); break;
	case 1652: LAUNCH_KERNEL(1652); break;
	case 1653: LAUNCH_KERNEL(1653); break;
	case 1654: LAUNCH_KERNEL(1654); break;
	case 1655: LAUNCH_KERNEL(1655); break;
	case 1656: LAUNCH_KERNEL(1656); break;
	case 1657: LAUNCH_KERNEL(1657); break;
	case 1658: LAUNCH_KERNEL(1658); break;
	case 1659: LAUNCH_KERNEL(1659); break;
	case 1660: LAUNCH_KERNEL(1660); break;
	case 1661: LAUNCH_KERNEL(1661); break;
	case 1662: LAUNCH_KERNEL(1662); break;
	case 1663: LAUNCH_KERNEL(1663); break;
	case 1664: LAUNCH_KERNEL(1664); break;
	case 1665: LAUNCH_KERNEL(1665); break;
	case 1666: LAUNCH_KERNEL(1666); break;
	case 1667: LAUNCH_KERNEL(1667); break;
	case 1668: LAUNCH_KERNEL(1668); break;
	case 1669: LAUNCH_KERNEL(1669); break;
	case 1670: LAUNCH_KERNEL(1670); break;
	case 1671: LAUNCH_KERNEL(1671); break;
	case 1672: LAUNCH_KERNEL(1672); break;
	case 1673: LAUNCH_KERNEL(1673); break;
	case 1674: LAUNCH_KERNEL(1674); break;
	case 1675: LAUNCH_KERNEL(1675); break;
	case 1676: LAUNCH_KERNEL(1676); break;
	case 1677: LAUNCH_KERNEL(1677); break;
	case 1678: LAUNCH_KERNEL(1678); break;
	case 1679: LAUNCH_KERNEL(1679); break;
	case 1680: LAUNCH_KERNEL(1680); break;
	case 1681: LAUNCH_KERNEL(1681); break;
	case 1682: LAUNCH_KERNEL(1682); break;
	case 1683: LAUNCH_KERNEL(1683); break;
	case 1684: LAUNCH_KERNEL(1684); break;
	case 1685: LAUNCH_KERNEL(1685); break;
	case 1686: LAUNCH_KERNEL(1686); break;
	case 1687: LAUNCH_KERNEL(1687); break;
	case 1688: LAUNCH_KERNEL(1688); break;
	case 1689: LAUNCH_KERNEL(1689); break;
	case 1690: LAUNCH_KERNEL(1690); break;
	case 1691: LAUNCH_KERNEL(1691); break;
	case 1692: LAUNCH_KERNEL(1692); break;
	case 1693: LAUNCH_KERNEL(1693); break;
	case 1694: LAUNCH_KERNEL(1694); break;
	case 1695: LAUNCH_KERNEL(1695); break;
	case 1696: LAUNCH_KERNEL(1696); break;
	case 1697: LAUNCH_KERNEL(1697); break;
	case 1698: LAUNCH_KERNEL(1698); break;
	case 1699: LAUNCH_KERNEL(1699); break;
	case 1700: LAUNCH_KERNEL(1700); break;
	case 1701: LAUNCH_KERNEL(1701); break;
	case 1702: LAUNCH_KERNEL(1702); break;
	case 1703: LAUNCH_KERNEL(1703); break;
	case 1704: LAUNCH_KERNEL(1704); break;
	case 1705: LAUNCH_KERNEL(1705); break;
	case 1706: LAUNCH_KERNEL(1706); break;
	case 1707: LAUNCH_KERNEL(1707); break;
	case 1708: LAUNCH_KERNEL(1708); break;
	case 1709: LAUNCH_KERNEL(1709); break;
	case 1710: LAUNCH_KERNEL(1710); break;
	case 1711: LAUNCH_KERNEL(1711); break;
	case 1712: LAUNCH_KERNEL(1712); break;
	case 1713: LAUNCH_KERNEL(1713); break;
	case 1714: LAUNCH_KERNEL(1714); break;
	case 1715: LAUNCH_KERNEL(1715); break;
	case 1716: LAUNCH_KERNEL(1716); break;
	case 1717: LAUNCH_KERNEL(1717); break;
	case 1718: LAUNCH_KERNEL(1718); break;
	case 1719: LAUNCH_KERNEL(1719); break;
	case 1720: LAUNCH_KERNEL(1720); break;
	case 1721: LAUNCH_KERNEL(1721); break;
	case 1722: LAUNCH_KERNEL(1722); break;
	case 1723: LAUNCH_KERNEL(1723); break;
	case 1724: LAUNCH_KERNEL(1724); break;
	case 1725: LAUNCH_KERNEL(1725); break;
	case 1726: LAUNCH_KERNEL(1726); break;
	case 1727: LAUNCH_KERNEL(1727); break;
	case 1728: LAUNCH_KERNEL(1728); break;
	case 1729: LAUNCH_KERNEL(1729); break;
	case 1730: LAUNCH_KERNEL(1730); break;
	case 1731: LAUNCH_KERNEL(1731); break;
	case 1732: LAUNCH_KERNEL(1732); break;
	case 1733: LAUNCH_KERNEL(1733); break;
	case 1734: LAUNCH_KERNEL(1734); break;
	case 1735: LAUNCH_KERNEL(1735); break;
	case 1736: LAUNCH_KERNEL(1736); break;
	case 1737: LAUNCH_KERNEL(1737); break;
	case 1738: LAUNCH_KERNEL(1738); break;
	case 1739: LAUNCH_KERNEL(1739); break;
	case 1740: LAUNCH_KERNEL(1740); break;
	case 1741: LAUNCH_KERNEL(1741); break;
	case 1742: LAUNCH_KERNEL(1742); break;
	case 1743: LAUNCH_KERNEL(1743); break;
	case 1744: LAUNCH_KERNEL(1744); break;
	case 1745: LAUNCH_KERNEL(1745); break;
	case 1746: LAUNCH_KERNEL(1746); break;
	case 1747: LAUNCH_KERNEL(1747); break;
	case 1748: LAUNCH_KERNEL(1748); break;
	case 1749: LAUNCH_KERNEL(1749); break;
	case 1750: LAUNCH_KERNEL(1750); break;
	case 1751: LAUNCH_KERNEL(1751); break;
	case 1752: LAUNCH_KERNEL(1752); break;
	case 1753: LAUNCH_KERNEL(1753); break;
	case 1754: LAUNCH_KERNEL(1754); break;
	case 1755: LAUNCH_KERNEL(1755); break;
	case 1756: LAUNCH_KERNEL(1756); break;
	case 1757: LAUNCH_KERNEL(1757); break;
	case 1758: LAUNCH_KERNEL(1758); break;
	case 1759: LAUNCH_KERNEL(1759); break;
	case 1760: LAUNCH_KERNEL(1760); break;
	case 1761: LAUNCH_KERNEL(1761); break;
	case 1762: LAUNCH_KERNEL(1762); break;
	case 1763: LAUNCH_KERNEL(1763); break;
	case 1764: LAUNCH_KERNEL(1764); break;
	case 1765: LAUNCH_KERNEL(1765); break;
	case 1766: LAUNCH_KERNEL(1766); break;
	case 1767: LAUNCH_KERNEL(1767); break;
	case 1768: LAUNCH_KERNEL(1768); break;
	case 1769: LAUNCH_KERNEL(1769); break;
	case 1770: LAUNCH_KERNEL(1770); break;
	case 1771: LAUNCH_KERNEL(1771); break;
	case 1772: LAUNCH_KERNEL(1772); break;
	case 1773: LAUNCH_KERNEL(1773); break;
	case 1774: LAUNCH_KERNEL(1774); break;
	case 1775: LAUNCH_KERNEL(1775); break;
	case 1776: LAUNCH_KERNEL(1776); break;
	case 1777: LAUNCH_KERNEL(1777); break;
	case 1778: LAUNCH_KERNEL(1778); break;
	case 1779: LAUNCH_KERNEL(1779); break;
	case 1780: LAUNCH_KERNEL(1780); break;
	case 1781: LAUNCH_KERNEL(1781); break;
	case 1782: LAUNCH_KERNEL(1782); break;
	case 1783: LAUNCH_KERNEL(1783); break;
	case 1784: LAUNCH_KERNEL(1784); break;
	case 1785: LAUNCH_KERNEL(1785); break;
	case 1786: LAUNCH_KERNEL(1786); break;
	case 1787: LAUNCH_KERNEL(1787); break;
	case 1788: LAUNCH_KERNEL(1788); break;
	case 1789: LAUNCH_KERNEL(1789); break;
	case 1790: LAUNCH_KERNEL(1790); break;
	case 1791: LAUNCH_KERNEL(1791); break;
	case 1792: LAUNCH_KERNEL(1792); break;
	case 1793: LAUNCH_KERNEL(1793); break;
	case 1794: LAUNCH_KERNEL(1794); break;
	case 1795: LAUNCH_KERNEL(1795); break;
	case 1796: LAUNCH_KERNEL(1796); break;
	case 1797: LAUNCH_KERNEL(1797); break;
	case 1798: LAUNCH_KERNEL(1798); break;
	case 1799: LAUNCH_KERNEL(1799); break;
	case 1800: LAUNCH_KERNEL(1800); break;
	case 1801: LAUNCH_KERNEL(1801); break;
	case 1802: LAUNCH_KERNEL(1802); break;
	case 1803: LAUNCH_KERNEL(1803); break;
	case 1804: LAUNCH_KERNEL(1804); break;
	case 1805: LAUNCH_KERNEL(1805); break;
	case 1806: LAUNCH_KERNEL(1806); break;
	case 1807: LAUNCH_KERNEL(1807); break;
	case 1808: LAUNCH_KERNEL(1808); break;
	case 1809: LAUNCH_KERNEL(1809); break;
	case 1810: LAUNCH_KERNEL(1810); break;
	case 1811: LAUNCH_KERNEL(1811); break;
	case 1812: LAUNCH_KERNEL(1812); break;
	case 1813: LAUNCH_KERNEL(1813); break;
	case 1814: LAUNCH_KERNEL(1814); break;
	case 1815: LAUNCH_KERNEL(1815); break;
	case 1816: LAUNCH_KERNEL(1816); break;
	case 1817: LAUNCH_KERNEL(1817); break;
	case 1818: LAUNCH_KERNEL(1818); break;
	case 1819: LAUNCH_KERNEL(1819); break;
	case 1820: LAUNCH_KERNEL(1820); break;
	case 1821: LAUNCH_KERNEL(1821); break;
	case 1822: LAUNCH_KERNEL(1822); break;
	case 1823: LAUNCH_KERNEL(1823); break;
	case 1824: LAUNCH_KERNEL(1824); break;
	case 1825: LAUNCH_KERNEL(1825); break;
	case 1826: LAUNCH_KERNEL(1826); break;
	case 1827: LAUNCH_KERNEL(1827); break;
	case 1828: LAUNCH_KERNEL(1828); break;
	case 1829: LAUNCH_KERNEL(1829); break;
	case 1830: LAUNCH_KERNEL(1830); break;
	case 1831: LAUNCH_KERNEL(1831); break;
	case 1832: LAUNCH_KERNEL(1832); break;
	case 1833: LAUNCH_KERNEL(1833); break;
	case 1834: LAUNCH_KERNEL(1834); break;
	case 1835: LAUNCH_KERNEL(1835); break;
	case 1836: LAUNCH_KERNEL(1836); break;
	case 1837: LAUNCH_KERNEL(1837); break;
	case 1838: LAUNCH_KERNEL(1838); break;
	case 1839: LAUNCH_KERNEL(1839); break;
	case 1840: LAUNCH_KERNEL(1840); break;
	case 1841: LAUNCH_KERNEL(1841); break;
	case 1842: LAUNCH_KERNEL(1842); break;
	case 1843: LAUNCH_KERNEL(1843); break;
	case 1844: LAUNCH_KERNEL(1844); break;
	case 1845: LAUNCH_KERNEL(1845); break;
	case 1846: LAUNCH_KERNEL(1846); break;
	case 1847: LAUNCH_KERNEL(1847); break;
	case 1848: LAUNCH_KERNEL(1848); break;
	case 1849: LAUNCH_KERNEL(1849); break;
	case 1850: LAUNCH_KERNEL(1850); break;
	case 1851: LAUNCH_KERNEL(1851); break;
	case 1852: LAUNCH_KERNEL(1852); break;
	case 1853: LAUNCH_KERNEL(1853); break;
	case 1854: LAUNCH_KERNEL(1854); break;
	case 1855: LAUNCH_KERNEL(1855); break;
	case 1856: LAUNCH_KERNEL(1856); break;
	case 1857: LAUNCH_KERNEL(1857); break;
	case 1858: LAUNCH_KERNEL(1858); break;
	case 1859: LAUNCH_KERNEL(1859); break;
	case 1860: LAUNCH_KERNEL(1860); break;
	case 1861: LAUNCH_KERNEL(1861); break;
	case 1862: LAUNCH_KERNEL(1862); break;
	case 1863: LAUNCH_KERNEL(1863); break;
	case 1864: LAUNCH_KERNEL(1864); break;
	case 1865: LAUNCH_KERNEL(1865); break;
	case 1866: LAUNCH_KERNEL(1866); break;
	case 1867: LAUNCH_KERNEL(1867); break;
	case 1868: LAUNCH_KERNEL(1868); break;
	case 1869: LAUNCH_KERNEL(1869); break;
	case 1870: LAUNCH_KERNEL(1870); break;
	case 1871: LAUNCH_KERNEL(1871); break;
	case 1872: LAUNCH_KERNEL(1872); break;
	case 1873: LAUNCH_KERNEL(1873); break;
	case 1874: LAUNCH_KERNEL(1874); break;
	case 1875: LAUNCH_KERNEL(1875); break;
	case 1876: LAUNCH_KERNEL(1876); break;
	case 1877: LAUNCH_KERNEL(1877); break;
	case 1878: LAUNCH_KERNEL(1878); break;
	case 1879: LAUNCH_KERNEL(1879); break;
	case 1880: LAUNCH_KERNEL(1880); break;
	case 1881: LAUNCH_KERNEL(1881); break;
	case 1882: LAUNCH_KERNEL(1882); break;
	case 1883: LAUNCH_KERNEL(1883); break;
	case 1884: LAUNCH_KERNEL(1884); break;
	case 1885: LAUNCH_KERNEL(1885); break;
	case 1886: LAUNCH_KERNEL(1886); break;
	case 1887: LAUNCH_KERNEL(1887); break;
	case 1888: LAUNCH_KERNEL(1888); break;
	case 1889: LAUNCH_KERNEL(1889); break;
	case 1890: LAUNCH_KERNEL(1890); break;
	case 1891: LAUNCH_KERNEL(1891); break;
	case 1892: LAUNCH_KERNEL(1892); break;
	case 1893: LAUNCH_KERNEL(1893); break;
	case 1894: LAUNCH_KERNEL(1894); break;
	case 1895: LAUNCH_KERNEL(1895); break;
	case 1896: LAUNCH_KERNEL(1896); break;
	case 1897: LAUNCH_KERNEL(1897); break;
	case 1898: LAUNCH_KERNEL(1898); break;
	case 1899: LAUNCH_KERNEL(1899); break;
	case 1900: LAUNCH_KERNEL(1900); break;
	case 1901: LAUNCH_KERNEL(1901); break;
	case 1902: LAUNCH_KERNEL(1902); break;
	case 1903: LAUNCH_KERNEL(1903); break;
	case 1904: LAUNCH_KERNEL(1904); break;
	case 1905: LAUNCH_KERNEL(1905); break;
	case 1906: LAUNCH_KERNEL(1906); break;
	case 1907: LAUNCH_KERNEL(1907); break;
	case 1908: LAUNCH_KERNEL(1908); break;
	case 1909: LAUNCH_KERNEL(1909); break;
	case 1910: LAUNCH_KERNEL(1910); break;
	case 1911: LAUNCH_KERNEL(1911); break;
	case 1912: LAUNCH_KERNEL(1912); break;
	case 1913: LAUNCH_KERNEL(1913); break;
	case 1914: LAUNCH_KERNEL(1914); break;
	case 1915: LAUNCH_KERNEL(1915); break;
	case 1916: LAUNCH_KERNEL(1916); break;
	case 1917: LAUNCH_KERNEL(1917); break;
	case 1918: LAUNCH_KERNEL(1918); break;
	case 1919: LAUNCH_KERNEL(1919); break;
	case 1920: LAUNCH_KERNEL(1920); break;
	case 1921: LAUNCH_KERNEL(1921); break;
	case 1922: LAUNCH_KERNEL(1922); break;
	case 1923: LAUNCH_KERNEL(1923); break;
	case 1924: LAUNCH_KERNEL(1924); break;
	case 1925: LAUNCH_KERNEL(1925); break;
	case 1926: LAUNCH_KERNEL(1926); break;
	case 1927: LAUNCH_KERNEL(1927); break;
	case 1928: LAUNCH_KERNEL(1928); break;
	case 1929: LAUNCH_KERNEL(1929); break;
	case 1930: LAUNCH_KERNEL(1930); break;
	case 1931: LAUNCH_KERNEL(1931); break;
	case 1932: LAUNCH_KERNEL(1932); break;
	case 1933: LAUNCH_KERNEL(1933); break;
	case 1934: LAUNCH_KERNEL(1934); break;
	case 1935: LAUNCH_KERNEL(1935); break;
	case 1936: LAUNCH_KERNEL(1936); break;
	case 1937: LAUNCH_KERNEL(1937); break;
	case 1938: LAUNCH_KERNEL(1938); break;
	case 1939: LAUNCH_KERNEL(1939); break;
	case 1940: LAUNCH_KERNEL(1940); break;
	case 1941: LAUNCH_KERNEL(1941); break;
	case 1942: LAUNCH_KERNEL(1942); break;
	case 1943: LAUNCH_KERNEL(1943); break;
	case 1944: LAUNCH_KERNEL(1944); break;
	case 1945: LAUNCH_KERNEL(1945); break;
	case 1946: LAUNCH_KERNEL(1946); break;
	case 1947: LAUNCH_KERNEL(1947); break;
	case 1948: LAUNCH_KERNEL(1948); break;
	case 1949: LAUNCH_KERNEL(1949); break;
	case 1950: LAUNCH_KERNEL(1950); break;
	case 1951: LAUNCH_KERNEL(1951); break;
	case 1952: LAUNCH_KERNEL(1952); break;
	case 1953: LAUNCH_KERNEL(1953); break;
	case 1954: LAUNCH_KERNEL(1954); break;
	case 1955: LAUNCH_KERNEL(1955); break;
	case 1956: LAUNCH_KERNEL(1956); break;
	case 1957: LAUNCH_KERNEL(1957); break;
	case 1958: LAUNCH_KERNEL(1958); break;
	case 1959: LAUNCH_KERNEL(1959); break;
	case 1960: LAUNCH_KERNEL(1960); break;
	case 1961: LAUNCH_KERNEL(1961); break;
	case 1962: LAUNCH_KERNEL(1962); break;
	case 1963: LAUNCH_KERNEL(1963); break;
	case 1964: LAUNCH_KERNEL(1964); break;
	case 1965: LAUNCH_KERNEL(1965); break;
	case 1966: LAUNCH_KERNEL(1966); break;
	case 1967: LAUNCH_KERNEL(1967); break;
	case 1968: LAUNCH_KERNEL(1968); break;
	case 1969: LAUNCH_KERNEL(1969); break;
	case 1970: LAUNCH_KERNEL(1970); break;
	case 1971: LAUNCH_KERNEL(1971); break;
	case 1972: LAUNCH_KERNEL(1972); break;
	case 1973: LAUNCH_KERNEL(1973); break;
	case 1974: LAUNCH_KERNEL(1974); break;
	case 1975: LAUNCH_KERNEL(1975); break;
	case 1976: LAUNCH_KERNEL(1976); break;
	case 1977: LAUNCH_KERNEL(1977); break;
	case 1978: LAUNCH_KERNEL(1978); break;
	case 1979: LAUNCH_KERNEL(1979); break;
	case 1980: LAUNCH_KERNEL(1980); break;
	case 1981: LAUNCH_KERNEL(1981); break;
	case 1982: LAUNCH_KERNEL(1982); break;
	case 1983: LAUNCH_KERNEL(1983); break;
	case 1984: LAUNCH_KERNEL(1984); break;
	case 1985: LAUNCH_KERNEL(1985); break;
	case 1986: LAUNCH_KERNEL(1986); break;
	case 1987: LAUNCH_KERNEL(1987); break;
	case 1988: LAUNCH_KERNEL(1988); break;
	case 1989: LAUNCH_KERNEL(1989); break;
	case 1990: LAUNCH_KERNEL(1990); break;
	case 1991: LAUNCH_KERNEL(1991); break;
	case 1992: LAUNCH_KERNEL(1992); break;
	case 1993: LAUNCH_KERNEL(1993); break;
	case 1994: LAUNCH_KERNEL(1994); break;
	case 1995: LAUNCH_KERNEL(1995); break;
	case 1996: LAUNCH_KERNEL(1996); break;
	case 1997: LAUNCH_KERNEL(1997); break;
	case 1998: LAUNCH_KERNEL(1998); break;
	case 1999: LAUNCH_KERNEL(1999); break;
	case 2000: LAUNCH_KERNEL(2000); break;
	case 2001: LAUNCH_KERNEL(2001); break;
	case 2002: LAUNCH_KERNEL(2002); break;
	case 2003: LAUNCH_KERNEL(2003); break;
	case 2004: LAUNCH_KERNEL(2004); break;
	case 2005: LAUNCH_KERNEL(2005); break;
	case 2006: LAUNCH_KERNEL(2006); break;
	case 2007: LAUNCH_KERNEL(2007); break;
	case 2008: LAUNCH_KERNEL(2008); break;
	case 2009: LAUNCH_KERNEL(2009); break;
	case 2010: LAUNCH_KERNEL(2010); break;
	case 2011: LAUNCH_KERNEL(2011); break;
	case 2012: LAUNCH_KERNEL(2012); break;
	case 2013: LAUNCH_KERNEL(2013); break;
	case 2014: LAUNCH_KERNEL(2014); break;
	case 2015: LAUNCH_KERNEL(2015); break;
	case 2016: LAUNCH_KERNEL(2016); break;
	case 2017: LAUNCH_KERNEL(2017); break;
	case 2018: LAUNCH_KERNEL(2018); break;
	case 2019: LAUNCH_KERNEL(2019); break;
	case 2020: LAUNCH_KERNEL(2020); break;
	case 2021: LAUNCH_KERNEL(2021); break;
	case 2022: LAUNCH_KERNEL(2022); break;
	case 2023: LAUNCH_KERNEL(2023); break;
	case 2024: LAUNCH_KERNEL(2024); break;
	case 2025: LAUNCH_KERNEL(2025); break;
	case 2026: LAUNCH_KERNEL(2026); break;
	case 2027: LAUNCH_KERNEL(2027); break;
	case 2028: LAUNCH_KERNEL(2028); break;
	case 2029: LAUNCH_KERNEL(2029); break;
	case 2030: LAUNCH_KERNEL(2030); break;
	case 2031: LAUNCH_KERNEL(2031); break;
	case 2032: LAUNCH_KERNEL(2032); break;
	case 2033: LAUNCH_KERNEL(2033); break;
	case 2034: LAUNCH_KERNEL(2034); break;
	case 2035: LAUNCH_KERNEL(2035); break;
	case 2036: LAUNCH_KERNEL(2036); break;
	case 2037: LAUNCH_KERNEL(2037); break;
	case 2038: LAUNCH_KERNEL(2038); break;
	case 2039: LAUNCH_KERNEL(2039); break;
	case 2040: LAUNCH_KERNEL(2040); break;
	case 2041: LAUNCH_KERNEL(2041); break;
	case 2042: LAUNCH_KERNEL(2042); break;
	case 2043: LAUNCH_KERNEL(2043); break;
	case 2044: LAUNCH_KERNEL(2044); break;
	case 2045: LAUNCH_KERNEL(2045); break;
	case 2046: LAUNCH_KERNEL(2046); break;
	case 2047: LAUNCH_KERNEL(2047); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
