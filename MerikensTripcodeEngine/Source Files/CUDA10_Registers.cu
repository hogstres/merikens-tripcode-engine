// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

	

// The following is a heavy rewrite of DeepLearningJohnDoe's awesome Bitslice DES implementation
// for the NVIDIA Maxwell architecture. See:
// https://devtalk.nvidia.com/default/topic/860120/cuda-programming-and-performance/bitslice-des-optimization/4/
// https://github.com/DeepLearningJohnDoe/merikens-tripcode-engine/tree/PRV



// #define SINGLE_SALT

// 841.6M t/s (1 chunk, DEBUG_SALT_0, 10m)
// 833.9M t/s (1 chunk 2 streams, DEBUG_SALT_0, 5h)
// 835.2M t/s (1 chunks, 4096 kernels, 15m)
// 831.4M t/s (1 chunks, JD, 3m)

// 790.0M t/s (10000 chunks, DEBUG_SALT_0, 25m)
// 795.0M t/s (10000 chunks, JD, 27m)
// 787.1M t/s (10000 chunks, 4096 kernels, 11m)


///////////////////////////////////////////////////////////////////////////////
// INCLUDE FILE(S)                                                           //
///////////////////////////////////////////////////////////////////////////////

#include "MerikensTripcodeEngine.h"
#include "CUDA10_Registers_Kernel_Common.h"
#ifdef DEBUG_SALT_0
#define SALT 0
#endif
#include "CUDA10_Registers_Kernel.h"



///////////////////////////////////////////////////////////////////////////////
// CUDA SEARCH THREAD FOR 10 CHARACTER TRIPCODES                             //
///////////////////////////////////////////////////////////////////////////////

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#define CUDA_DES_DECLARE_KERNEL_LAUNCHER(n) \
	extern void CUDA_DES_InitializeKernelLauncher##n();\
	extern void CUDA_DES_LaunchKernel##n(\
		uint32_t numBlocksPerGrid,\
		hipDeviceProp_t CUDADeviceProperties,\
		hipStream_t currentStream,\
		unsigned char *cudaPassCountArray,\
		unsigned char *cudaTripcodeIndexArray,\
		uint32_t *cudaTripcodeChunkArray,\
		uint32_t numTripcodeChunk,\
		int32_t intSalt,\
		unsigned char *cudaKey0Array,\
		unsigned char *cudaKey7Array,\
		DES_Vector *cudaKeyVectorsFrom49To55,\
		unsigned char *cudaKeyAndRandomBytes,\
		int32_t searchMode)\

#define CUDA_DES_CALL_KERNEL_LAUNCHER(n) \
	CUDA_DES_LaunchKernel##n(\
		numBlocksPerGrid,\
		CUDADeviceProperties,\
		currentStream,\
		cudaPassCountArray,\
		cudaTripcodeIndexArray,\
		cudaTripcodeChunkArray,\
		numTripcodeChunk,\
		intSalt,\
		cudaKey0Array,\
		cudaKey7Array,\
		cudaKeyVectorsFrom49To55,\
		cudaKeyAndRandomBytes,\
		searchMode)\

CUDA_DES_DECLARE_KERNEL_LAUNCHER(0);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(1);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(2);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(3);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(4);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(5);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(6);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(7);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(8);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(9);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(10);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(11);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(12);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(13);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(14);
CUDA_DES_DECLARE_KERNEL_LAUNCHER(15);

#endif



#define SET_BIT_FOR_KEY7(var, k) if (key7 & (0x1 << (k))) (var) |= 0x1 << tripcodeIndex

unsigned WINAPI Thread_SearchForDESTripcodesOnCUDADevice_Registers(LPVOID info)
{
	hipDeviceProp_t  CUDADeviceProperties;
	uint32_t    numBlocksPerSM;
	uint32_t    numBlocksPerGrid;
	unsigned char  *passCountArray = NULL;
	unsigned char  *cudaPassCountArray = NULL;
	unsigned char  *tripcodeIndexArray = NULL;
	unsigned char  *cudaTripcodeIndexArray = NULL;
	uint32_t   *cudaTripcodeChunkArray = NULL;
	unsigned char  *cudaKey0Array = NULL;
	unsigned char  *cudaKey7Array = NULL;
	unsigned char  *cudaKeyAndRandomBytes = NULL;
	DES_Vector     *cudaKeyVectorsFrom49To55;
	unsigned char   key0Array[CUDA_DES_MAX_PASS_COUNT];
	unsigned char   key7Array[CUDA_DES_BS_DEPTH * 2];
	unsigned char   keyAndRandomBytes[MAX_LEN_TRIPCODE + 1];

	unsigned char  *prevPassCountArray = NULL;
	unsigned char  *cudaPrevPassCountArray = NULL;
	unsigned char  *prevTripcodeIndexArray = NULL;
	unsigned char  *cudaPrevTripcodeIndexArray = NULL;
	unsigned char   prevKey0Array[CUDA_DES_MAX_PASS_COUNT];
	unsigned char   prevKey7Array[CUDA_DES_BS_DEPTH * 2];
	unsigned char   prevKeyAndRandomBytes[MAX_LEN_TRIPCODE + 1];

	uint32_t    numThreadsPerGrid;
	unsigned char   salt[3];
	char            status[LEN_LINE_BUFFER_FOR_SCREEN] = "";
	double          timeElapsed = 0;
	double          numGeneratedTripcodes = 0;
	double          speed = 0;
	DWORD           startingTime;
	DWORD           endingTime;
	double          deltaTime;

	keyAndRandomBytes[lenTripcode] = '\0';
	salt[2] = '\0';
	
	CUDA_ERROR(hipSetDevice(((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	CUDA_ERROR(hipGetDeviceProperties(&CUDADeviceProperties, ((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	if (CUDADeviceProperties.computeMode == hipComputeModeProhibited) {
		sprintf(status, "[disabled]");
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), status);
		return 0;
	}

	numBlocksPerSM = options.CUDANumBlocksPerSM;
	numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
	numThreadsPerGrid = CUDA_DES_NUM_THREADS_PER_BLOCK * numBlocksPerGrid;

	CUDA_ERROR(hipMalloc((void **)&cudaTripcodeChunkArray,   sizeof(uint32_t) * numTripcodeChunk)); 
	CUDA_ERROR(hipMalloc((void **)&cudaKey0Array,            sizeof(unsigned char) * CUDA_DES_MAX_PASS_COUNT)); 
	CUDA_ERROR(hipMalloc((void **)&cudaKey7Array,            sizeof(unsigned char) * CUDA_DES_BS_DEPTH * 2)); 
	CUDA_ERROR(hipMalloc((void **)&cudaKeyVectorsFrom49To55, sizeof(DES_Vector) * 7 * 2)); 
	CUDA_ERROR(hipMalloc((void **)&cudaKeyAndRandomBytes,    sizeof(unsigned char) * 8)); 
	
	(((CUDADeviceSearchThreadInfo *)info)->criticalSection).lock();
#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE
	CUDA_DES_InitializeKernelLauncher0();
	CUDA_DES_InitializeKernelLauncher1();
	CUDA_DES_InitializeKernelLauncher2();
	CUDA_DES_InitializeKernelLauncher3();
	CUDA_DES_InitializeKernelLauncher4();
	CUDA_DES_InitializeKernelLauncher5();
	CUDA_DES_InitializeKernelLauncher6();
	CUDA_DES_InitializeKernelLauncher7();
	CUDA_DES_InitializeKernelLauncher8();
	CUDA_DES_InitializeKernelLauncher9();
	CUDA_DES_InitializeKernelLauncher10();
	CUDA_DES_InitializeKernelLauncher11();
	CUDA_DES_InitializeKernelLauncher12();
	CUDA_DES_InitializeKernelLauncher13();
	CUDA_DES_InitializeKernelLauncher14();
	CUDA_DES_InitializeKernelLauncher15();
#endif
	CUDA_ERROR(hipMemcpy(cudaTripcodeChunkArray, tripcodeChunkArray, sizeof(uint32_t) * numTripcodeChunk, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),  keyCharTable_FirstByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte), keyCharTable_SecondByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),  compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
	(((CUDADeviceSearchThreadInfo *)info)->criticalSection).unlock();
		
	startingTime = timeGetTime();

	hipStream_t currentStream;
	CUDA_ERROR(hipStreamCreate(&currentStream));
	BOOL prevDataExists = FALSE;
	passCountArray         = (unsigned char *)malloc(sizeof(unsigned char) * numThreadsPerGrid); ERROR0(passCountArray         == NULL, ERROR_NO_MEMORY, GetErrorMessage(ERROR_NO_MEMORY));
	tripcodeIndexArray     = (unsigned char *)malloc(sizeof(unsigned char) * numThreadsPerGrid); ERROR0(tripcodeIndexArray     == NULL, ERROR_NO_MEMORY, GetErrorMessage(ERROR_NO_MEMORY));
	prevPassCountArray     = (unsigned char *)malloc(sizeof(unsigned char) * numThreadsPerGrid); ERROR0(prevPassCountArray     == NULL, ERROR_NO_MEMORY, GetErrorMessage(ERROR_NO_MEMORY));
	prevTripcodeIndexArray = (unsigned char *)malloc(sizeof(unsigned char) * numThreadsPerGrid); ERROR0(prevTripcodeIndexArray == NULL, ERROR_NO_MEMORY, GetErrorMessage(ERROR_NO_MEMORY));
	CUDA_ERROR(hipMalloc((void **)&cudaPassCountArray,           sizeof(unsigned char) * numThreadsPerGrid));
	CUDA_ERROR(hipMalloc((void **)&cudaTripcodeIndexArray,       sizeof(unsigned char) * numThreadsPerGrid));
	CUDA_ERROR(hipMalloc((void **)&cudaPrevPassCountArray,       sizeof(unsigned char) * numThreadsPerGrid));
	CUDA_ERROR(hipMalloc((void **)&cudaPrevTripcodeIndexArray,   sizeof(unsigned char) * numThreadsPerGrid));
	while (!GetTerminationState()) {
		// Choose the first 3 characters of the keyAndRandomBytes.
		int32_t intSalt;
		for (int32_t i = 3; i < lenTripcode; ++i)
			keyAndRandomBytes[i] = 'A';
		do {
			SetCharactersInTripcodeKey(keyAndRandomBytes, 3);
			salt[0] = CONVERT_CHAR_FOR_SALT(keyAndRandomBytes[1]);
			salt[1] = CONVERT_CHAR_FOR_SALT(keyAndRandomBytes[2]);
			intSalt = charToIndexTableForDES[salt[0]] | (charToIndexTableForDES[salt[1]] << 6);
		} while (
#ifdef SINGLE_SALT
                    intSalt || 
#endif
				    !IsValidKey(keyAndRandomBytes));

		//
		unsigned char randomByteForKey0 = RandomByte();
		int32_t j = 0;
		for (int32_t i = 0; i < CUDA_DES_MAX_PASS_COUNT; ++i) {
			do {
				keyAndRandomBytes[0] = keyCharTable_FirstByte[randomByteForKey0 + j++];
			} while(!IsValidKey(keyAndRandomBytes));
			key0Array[i] = keyAndRandomBytes[0];
		}
		
		// Generate random bytes for the key to ensure its randomness.
		for (int32_t i = 3; i < lenTripcode; ++i)
			keyAndRandomBytes[i] = RandomByte();

		//
		DES_Vector  keyVectorsFrom49To55[7 * 2] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
		for (int32_t tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {
			unsigned char key7 = key7Array[tripcodeIndex] = keyCharTable_FirstByte[keyAndRandomBytes[7] + tripcodeIndex];
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[0], 0);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[1], 1);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[2], 2);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[3], 3);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[4], 4);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[5], 5);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[6], 6);
			key7 = key7Array[tripcodeIndex + CUDA_DES_BS_DEPTH] = keyCharTable_SecondByte[keyAndRandomBytes[7] + tripcodeIndex];
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[0 + 7], 0);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[1 + 7], 1);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[2 + 7], 2);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[3 + 7], 3);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[4 + 7], 4);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[5 + 7], 5);
			SET_BIT_FOR_KEY7(keyVectorsFrom49To55[6 + 7], 6);
		}

		// Call an appropriate CUDA kernel.
		CUDA_ERROR(hipMemcpyAsync(cudaKey0Array, key0Array, sizeof(key0Array), hipMemcpyHostToDevice, currentStream));
		CUDA_ERROR(hipMemcpyAsync(cudaKey7Array, key7Array, sizeof(key7Array), hipMemcpyHostToDevice, currentStream));
		CUDA_ERROR(hipMemcpyAsync(cudaKeyVectorsFrom49To55, keyVectorsFrom49To55, sizeof(keyVectorsFrom49To55), hipMemcpyHostToDevice, currentStream))
		CUDA_ERROR(hipMemcpyAsync(cudaKeyAndRandomBytes, keyAndRandomBytes, 8, hipMemcpyHostToDevice, currentStream));
#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE
		switch (intSalt / 256) {
		case 0: CUDA_DES_CALL_KERNEL_LAUNCHER(0); break;
		case 1: CUDA_DES_CALL_KERNEL_LAUNCHER(1); break;
		case 2: CUDA_DES_CALL_KERNEL_LAUNCHER(2); break;
		case 3: CUDA_DES_CALL_KERNEL_LAUNCHER(3); break;
		case 4: CUDA_DES_CALL_KERNEL_LAUNCHER(4); break;
		case 5: CUDA_DES_CALL_KERNEL_LAUNCHER(5); break;
		case 6: CUDA_DES_CALL_KERNEL_LAUNCHER(6); break;
		case 7: CUDA_DES_CALL_KERNEL_LAUNCHER(7); break;
		case 8: CUDA_DES_CALL_KERNEL_LAUNCHER(8); break;
		case 9: CUDA_DES_CALL_KERNEL_LAUNCHER(9); break;
		case 10: CUDA_DES_CALL_KERNEL_LAUNCHER(10); break;
		case 11: CUDA_DES_CALL_KERNEL_LAUNCHER(11); break;
		case 12: CUDA_DES_CALL_KERNEL_LAUNCHER(12); break;
		case 13: CUDA_DES_CALL_KERNEL_LAUNCHER(13); break;
		case 14: CUDA_DES_CALL_KERNEL_LAUNCHER(14); break;
		case 15: CUDA_DES_CALL_KERNEL_LAUNCHER(15); break;
		default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
		}
#else
		dim3 dimGrid(numBlocksPerGrid);
		dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
#ifdef DEBUG_SALT_0
		CUDA_DES_PerformSearch_0<<<dimGrid, dimBlock, 0, currentStream>>>(
#else
		CUDA_DES_PerformSearch<<<dimGrid, dimBlock, 0, currentStream>>>(
#endif	
			cudaPassCountArray,
			cudaTripcodeIndexArray,
			cudaTripcodeChunkArray,
			numTripcodeChunk,
			intSalt,
			cudaKey0Array,
			cudaKey7Array,
			cudaKeyVectorsFrom49To55,
			cudaKeyAndRandomBytes,
			searchMode);
#endif
		CUDA_ERROR(hipGetLastError());
		CUDA_ERROR(hipMemcpyAsync(passCountArray,     cudaPassCountArray,     sizeof(unsigned char) * numThreadsPerGrid, hipMemcpyDeviceToHost, currentStream));
		CUDA_ERROR(hipMemcpyAsync(tripcodeIndexArray, cudaTripcodeIndexArray, sizeof(unsigned char) * numThreadsPerGrid, hipMemcpyDeviceToHost, currentStream));

		// Process the output.
		TripcodeKeyPair tripcodes[32];
		int32_t numTripcodes = 0;
		if (prevDataExists) {
			for (int32_t i = 0; i < numThreadsPerGrid; i++){
				if (prevPassCountArray[i] < CUDA_DES_MAX_PASS_COUNT) {
					unsigned char key[MAX_LEN_TRIPCODE_KEY + 1];
					key[0] = prevKey0Array[prevPassCountArray[i]];
					key[1] = prevKeyAndRandomBytes[1];
					key[2] = prevKeyAndRandomBytes[2];
	
					BOOL isSecondByte =    ( IS_FIRST_BYTE_SJIS_FULL(prevKey0Array[0])                                                       && IS_FIRST_BYTE_SJIS_FULL(prevKeyAndRandomBytes[2]))
										|| (!IS_FIRST_BYTE_SJIS_FULL(prevKey0Array[0]) && !IS_FIRST_BYTE_SJIS_FULL(prevKeyAndRandomBytes[1]) && IS_FIRST_BYTE_SJIS_FULL(prevKeyAndRandomBytes[2]));
					int32_t threadIndex = i % CUDA_DES_NUM_THREADS_PER_BLOCK;
					int32_t blockIndex  = i / CUDA_DES_NUM_THREADS_PER_BLOCK;
					SET_KEY_CHAR(key[3], isSecondByte, keyCharTable_FirstByte, prevKeyAndRandomBytes[3] + (((threadIndex >> 6) &  7) | (((blockIndex  >> 12) & 7) << 3)));
					SET_KEY_CHAR(key[4], isSecondByte, keyCharTable_FirstByte, prevKeyAndRandomBytes[4] + ( (blockIndex  >> 6) & 63));
					SET_KEY_CHAR(key[5], isSecondByte, keyCharTable_FirstByte, prevKeyAndRandomBytes[5] + (  blockIndex        & 63));
					SET_KEY_CHAR(key[6], isSecondByte, keyCharTable_FirstByte, prevKeyAndRandomBytes[6] + (  threadIndex       & 63));
					key[7] = prevKey7Array[prevTripcodeIndexArray[i] + (isSecondByte ? CUDA_DES_BS_DEPTH : 0)];
					CreateKey8AndKey9(key);
					key[10] = '\0';
					strcpy((char *)tripcodes[numTripcodes].key.c, (char *)key);
					++numTripcodes;
				}
				if (numTripcodes > 0 && (numTripcodes >= sizeof(tripcodes) / sizeof(TripcodeKeyPair) || i >= numThreadsPerGrid - 1)) {
					Generate10CharTripcodes(tripcodes, numTripcodes);
					for (int32_t j = 0; j < numTripcodes; j++){
						ERROR0(!IsTripcodeChunkValid(tripcodes[j].tripcode.c), 
							   ERROR_TRIPCODE_VERIFICATION_FAILED, 
							   GetErrorMessage(ERROR_TRIPCODE_VERIFICATION_FAILED));
						ProcessPossibleMatch(tripcodes[j].tripcode.c, tripcodes[j].key.c);
					}
					numTripcodes = 0;
				}
			}
		}
		CUDA_ERROR(hipStreamSynchronize(currentStream));
		uint32_t numGeneratedTripcodesThisTime = 0;
		for (int32_t i = 0; i < numThreadsPerGrid; i++)
			numGeneratedTripcodesThisTime += CUDA_DES_BS_DEPTH * passCountArray[i];
		AddToNumGeneratedTripcodesByGPU(numGeneratedTripcodesThisTime);
		numGeneratedTripcodes += numGeneratedTripcodesThisTime;
#undef  SWAP
#define SWAP(t, a, b) { t temp; temp = (a); (a) = (b); (b) = temp; }
		SWAP(unsigned char *, passCountArray, prevPassCountArray);
		SWAP(unsigned char *, tripcodeIndexArray, prevTripcodeIndexArray);
		SWAP(unsigned char *, cudaPassCountArray, cudaPrevPassCountArray);
		SWAP(unsigned char *, cudaTripcodeIndexArray, cudaPrevTripcodeIndexArray);
		memcpy(prevKey0Array, key0Array, sizeof(key0Array));
		memcpy(prevKey7Array, key7Array, sizeof(key7Array));
		memcpy(prevKeyAndRandomBytes, keyAndRandomBytes, sizeof(keyAndRandomBytes));
		prevDataExists = TRUE;

		//
		endingTime = timeGetTime();
		deltaTime = (endingTime >= startingTime)
						? ((double)endingTime - (double)startingTime                     ) * 0.001
						: ((double)endingTime - (double)startingTime + (double)0xffffffff) * 0.001;
		while (GetPauseState() && !GetTerminationState())
			Sleep(PAUSE_INTERVAL);
		startingTime = timeGetTime();
		timeElapsed += deltaTime;
		speed = numGeneratedTripcodes / timeElapsed;
		sprintf(status,
			    "%.1lfM TPS, %d blocks/SM",
				speed / 1000000,
				numBlocksPerSM);
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), status);
	}

	RELEASE_AND_SET_TO_NULL(passCountArray,               free);
	RELEASE_AND_SET_TO_NULL(tripcodeIndexArray,           free);
	RELEASE_AND_SET_TO_NULL(cudaPassCountArray,           hipFree);
	RELEASE_AND_SET_TO_NULL(cudaTripcodeIndexArray,       hipFree);
	RELEASE_AND_SET_TO_NULL(prevPassCountArray,           free);
	RELEASE_AND_SET_TO_NULL(prevTripcodeIndexArray,       free);
	RELEASE_AND_SET_TO_NULL(cudaPrevPassCountArray,       hipFree);
	RELEASE_AND_SET_TO_NULL(cudaPrevTripcodeIndexArray,   hipFree);
	RELEASE_AND_SET_TO_NULL(cudaTripcodeChunkArray,   hipFree);
	RELEASE_AND_SET_TO_NULL(cudaKey0Array,            hipFree);
	RELEASE_AND_SET_TO_NULL(cudaKey7Array,            hipFree);
	RELEASE_AND_SET_TO_NULL(cudaKeyVectorsFrom49To55, hipFree);
	RELEASE_AND_SET_TO_NULL(cudaKeyAndRandomBytes,    hipFree);
	CUDA_ERROR(hipStreamDestroy(currentStream));

	return 0;
}
