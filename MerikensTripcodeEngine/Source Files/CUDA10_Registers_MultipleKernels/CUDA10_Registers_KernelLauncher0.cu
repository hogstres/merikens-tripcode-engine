// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 0
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4
#include "../CUDA10_Registers_Kernel.h"
#define SALT 5
#include "../CUDA10_Registers_Kernel.h"
#define SALT 6
#include "../CUDA10_Registers_Kernel.h"
#define SALT 7
#include "../CUDA10_Registers_Kernel.h"
#define SALT 8
#include "../CUDA10_Registers_Kernel.h"
#define SALT 9
#include "../CUDA10_Registers_Kernel.h"
#define SALT 10
#include "../CUDA10_Registers_Kernel.h"
#define SALT 11
#include "../CUDA10_Registers_Kernel.h"
#define SALT 12
#include "../CUDA10_Registers_Kernel.h"
#define SALT 13
#include "../CUDA10_Registers_Kernel.h"
#define SALT 14
#include "../CUDA10_Registers_Kernel.h"
#define SALT 15
#include "../CUDA10_Registers_Kernel.h"
#define SALT 16
#include "../CUDA10_Registers_Kernel.h"
#define SALT 17
#include "../CUDA10_Registers_Kernel.h"
#define SALT 18
#include "../CUDA10_Registers_Kernel.h"
#define SALT 19
#include "../CUDA10_Registers_Kernel.h"
#define SALT 20
#include "../CUDA10_Registers_Kernel.h"
#define SALT 21
#include "../CUDA10_Registers_Kernel.h"
#define SALT 22
#include "../CUDA10_Registers_Kernel.h"
#define SALT 23
#include "../CUDA10_Registers_Kernel.h"
#define SALT 24
#include "../CUDA10_Registers_Kernel.h"
#define SALT 25
#include "../CUDA10_Registers_Kernel.h"
#define SALT 26
#include "../CUDA10_Registers_Kernel.h"
#define SALT 27
#include "../CUDA10_Registers_Kernel.h"
#define SALT 28
#include "../CUDA10_Registers_Kernel.h"
#define SALT 29
#include "../CUDA10_Registers_Kernel.h"
#define SALT 30
#include "../CUDA10_Registers_Kernel.h"
#define SALT 31
#include "../CUDA10_Registers_Kernel.h"
#define SALT 32
#include "../CUDA10_Registers_Kernel.h"
#define SALT 33
#include "../CUDA10_Registers_Kernel.h"
#define SALT 34
#include "../CUDA10_Registers_Kernel.h"
#define SALT 35
#include "../CUDA10_Registers_Kernel.h"
#define SALT 36
#include "../CUDA10_Registers_Kernel.h"
#define SALT 37
#include "../CUDA10_Registers_Kernel.h"
#define SALT 38
#include "../CUDA10_Registers_Kernel.h"
#define SALT 39
#include "../CUDA10_Registers_Kernel.h"
#define SALT 40
#include "../CUDA10_Registers_Kernel.h"
#define SALT 41
#include "../CUDA10_Registers_Kernel.h"
#define SALT 42
#include "../CUDA10_Registers_Kernel.h"
#define SALT 43
#include "../CUDA10_Registers_Kernel.h"
#define SALT 44
#include "../CUDA10_Registers_Kernel.h"
#define SALT 45
#include "../CUDA10_Registers_Kernel.h"
#define SALT 46
#include "../CUDA10_Registers_Kernel.h"
#define SALT 47
#include "../CUDA10_Registers_Kernel.h"
#define SALT 48
#include "../CUDA10_Registers_Kernel.h"
#define SALT 49
#include "../CUDA10_Registers_Kernel.h"
#define SALT 50
#include "../CUDA10_Registers_Kernel.h"
#define SALT 51
#include "../CUDA10_Registers_Kernel.h"
#define SALT 52
#include "../CUDA10_Registers_Kernel.h"
#define SALT 53
#include "../CUDA10_Registers_Kernel.h"
#define SALT 54
#include "../CUDA10_Registers_Kernel.h"
#define SALT 55
#include "../CUDA10_Registers_Kernel.h"
#define SALT 56
#include "../CUDA10_Registers_Kernel.h"
#define SALT 57
#include "../CUDA10_Registers_Kernel.h"
#define SALT 58
#include "../CUDA10_Registers_Kernel.h"
#define SALT 59
#include "../CUDA10_Registers_Kernel.h"
#define SALT 60
#include "../CUDA10_Registers_Kernel.h"
#define SALT 61
#include "../CUDA10_Registers_Kernel.h"
#define SALT 62
#include "../CUDA10_Registers_Kernel.h"
#define SALT 63
#include "../CUDA10_Registers_Kernel.h"
#define SALT 64
#include "../CUDA10_Registers_Kernel.h"
#define SALT 65
#include "../CUDA10_Registers_Kernel.h"
#define SALT 66
#include "../CUDA10_Registers_Kernel.h"
#define SALT 67
#include "../CUDA10_Registers_Kernel.h"
#define SALT 68
#include "../CUDA10_Registers_Kernel.h"
#define SALT 69
#include "../CUDA10_Registers_Kernel.h"
#define SALT 70
#include "../CUDA10_Registers_Kernel.h"
#define SALT 71
#include "../CUDA10_Registers_Kernel.h"
#define SALT 72
#include "../CUDA10_Registers_Kernel.h"
#define SALT 73
#include "../CUDA10_Registers_Kernel.h"
#define SALT 74
#include "../CUDA10_Registers_Kernel.h"
#define SALT 75
#include "../CUDA10_Registers_Kernel.h"
#define SALT 76
#include "../CUDA10_Registers_Kernel.h"
#define SALT 77
#include "../CUDA10_Registers_Kernel.h"
#define SALT 78
#include "../CUDA10_Registers_Kernel.h"
#define SALT 79
#include "../CUDA10_Registers_Kernel.h"
#define SALT 80
#include "../CUDA10_Registers_Kernel.h"
#define SALT 81
#include "../CUDA10_Registers_Kernel.h"
#define SALT 82
#include "../CUDA10_Registers_Kernel.h"
#define SALT 83
#include "../CUDA10_Registers_Kernel.h"
#define SALT 84
#include "../CUDA10_Registers_Kernel.h"
#define SALT 85
#include "../CUDA10_Registers_Kernel.h"
#define SALT 86
#include "../CUDA10_Registers_Kernel.h"
#define SALT 87
#include "../CUDA10_Registers_Kernel.h"
#define SALT 88
#include "../CUDA10_Registers_Kernel.h"
#define SALT 89
#include "../CUDA10_Registers_Kernel.h"
#define SALT 90
#include "../CUDA10_Registers_Kernel.h"
#define SALT 91
#include "../CUDA10_Registers_Kernel.h"
#define SALT 92
#include "../CUDA10_Registers_Kernel.h"
#define SALT 93
#include "../CUDA10_Registers_Kernel.h"
#define SALT 94
#include "../CUDA10_Registers_Kernel.h"
#define SALT 95
#include "../CUDA10_Registers_Kernel.h"
#define SALT 96
#include "../CUDA10_Registers_Kernel.h"
#define SALT 97
#include "../CUDA10_Registers_Kernel.h"
#define SALT 98
#include "../CUDA10_Registers_Kernel.h"
#define SALT 99
#include "../CUDA10_Registers_Kernel.h"
#define SALT 100
#include "../CUDA10_Registers_Kernel.h"
#define SALT 101
#include "../CUDA10_Registers_Kernel.h"
#define SALT 102
#include "../CUDA10_Registers_Kernel.h"
#define SALT 103
#include "../CUDA10_Registers_Kernel.h"
#define SALT 104
#include "../CUDA10_Registers_Kernel.h"
#define SALT 105
#include "../CUDA10_Registers_Kernel.h"
#define SALT 106
#include "../CUDA10_Registers_Kernel.h"
#define SALT 107
#include "../CUDA10_Registers_Kernel.h"
#define SALT 108
#include "../CUDA10_Registers_Kernel.h"
#define SALT 109
#include "../CUDA10_Registers_Kernel.h"
#define SALT 110
#include "../CUDA10_Registers_Kernel.h"
#define SALT 111
#include "../CUDA10_Registers_Kernel.h"
#define SALT 112
#include "../CUDA10_Registers_Kernel.h"
#define SALT 113
#include "../CUDA10_Registers_Kernel.h"
#define SALT 114
#include "../CUDA10_Registers_Kernel.h"
#define SALT 115
#include "../CUDA10_Registers_Kernel.h"
#define SALT 116
#include "../CUDA10_Registers_Kernel.h"
#define SALT 117
#include "../CUDA10_Registers_Kernel.h"
#define SALT 118
#include "../CUDA10_Registers_Kernel.h"
#define SALT 119
#include "../CUDA10_Registers_Kernel.h"
#define SALT 120
#include "../CUDA10_Registers_Kernel.h"
#define SALT 121
#include "../CUDA10_Registers_Kernel.h"
#define SALT 122
#include "../CUDA10_Registers_Kernel.h"
#define SALT 123
#include "../CUDA10_Registers_Kernel.h"
#define SALT 124
#include "../CUDA10_Registers_Kernel.h"
#define SALT 125
#include "../CUDA10_Registers_Kernel.h"
#define SALT 126
#include "../CUDA10_Registers_Kernel.h"
#define SALT 127
#include "../CUDA10_Registers_Kernel.h"
#define SALT 128
#include "../CUDA10_Registers_Kernel.h"
#define SALT 129
#include "../CUDA10_Registers_Kernel.h"
#define SALT 130
#include "../CUDA10_Registers_Kernel.h"
#define SALT 131
#include "../CUDA10_Registers_Kernel.h"
#define SALT 132
#include "../CUDA10_Registers_Kernel.h"
#define SALT 133
#include "../CUDA10_Registers_Kernel.h"
#define SALT 134
#include "../CUDA10_Registers_Kernel.h"
#define SALT 135
#include "../CUDA10_Registers_Kernel.h"
#define SALT 136
#include "../CUDA10_Registers_Kernel.h"
#define SALT 137
#include "../CUDA10_Registers_Kernel.h"
#define SALT 138
#include "../CUDA10_Registers_Kernel.h"
#define SALT 139
#include "../CUDA10_Registers_Kernel.h"
#define SALT 140
#include "../CUDA10_Registers_Kernel.h"
#define SALT 141
#include "../CUDA10_Registers_Kernel.h"
#define SALT 142
#include "../CUDA10_Registers_Kernel.h"
#define SALT 143
#include "../CUDA10_Registers_Kernel.h"
#define SALT 144
#include "../CUDA10_Registers_Kernel.h"
#define SALT 145
#include "../CUDA10_Registers_Kernel.h"
#define SALT 146
#include "../CUDA10_Registers_Kernel.h"
#define SALT 147
#include "../CUDA10_Registers_Kernel.h"
#define SALT 148
#include "../CUDA10_Registers_Kernel.h"
#define SALT 149
#include "../CUDA10_Registers_Kernel.h"
#define SALT 150
#include "../CUDA10_Registers_Kernel.h"
#define SALT 151
#include "../CUDA10_Registers_Kernel.h"
#define SALT 152
#include "../CUDA10_Registers_Kernel.h"
#define SALT 153
#include "../CUDA10_Registers_Kernel.h"
#define SALT 154
#include "../CUDA10_Registers_Kernel.h"
#define SALT 155
#include "../CUDA10_Registers_Kernel.h"
#define SALT 156
#include "../CUDA10_Registers_Kernel.h"
#define SALT 157
#include "../CUDA10_Registers_Kernel.h"
#define SALT 158
#include "../CUDA10_Registers_Kernel.h"
#define SALT 159
#include "../CUDA10_Registers_Kernel.h"
#define SALT 160
#include "../CUDA10_Registers_Kernel.h"
#define SALT 161
#include "../CUDA10_Registers_Kernel.h"
#define SALT 162
#include "../CUDA10_Registers_Kernel.h"
#define SALT 163
#include "../CUDA10_Registers_Kernel.h"
#define SALT 164
#include "../CUDA10_Registers_Kernel.h"
#define SALT 165
#include "../CUDA10_Registers_Kernel.h"
#define SALT 166
#include "../CUDA10_Registers_Kernel.h"
#define SALT 167
#include "../CUDA10_Registers_Kernel.h"
#define SALT 168
#include "../CUDA10_Registers_Kernel.h"
#define SALT 169
#include "../CUDA10_Registers_Kernel.h"
#define SALT 170
#include "../CUDA10_Registers_Kernel.h"
#define SALT 171
#include "../CUDA10_Registers_Kernel.h"
#define SALT 172
#include "../CUDA10_Registers_Kernel.h"
#define SALT 173
#include "../CUDA10_Registers_Kernel.h"
#define SALT 174
#include "../CUDA10_Registers_Kernel.h"
#define SALT 175
#include "../CUDA10_Registers_Kernel.h"
#define SALT 176
#include "../CUDA10_Registers_Kernel.h"
#define SALT 177
#include "../CUDA10_Registers_Kernel.h"
#define SALT 178
#include "../CUDA10_Registers_Kernel.h"
#define SALT 179
#include "../CUDA10_Registers_Kernel.h"
#define SALT 180
#include "../CUDA10_Registers_Kernel.h"
#define SALT 181
#include "../CUDA10_Registers_Kernel.h"
#define SALT 182
#include "../CUDA10_Registers_Kernel.h"
#define SALT 183
#include "../CUDA10_Registers_Kernel.h"
#define SALT 184
#include "../CUDA10_Registers_Kernel.h"
#define SALT 185
#include "../CUDA10_Registers_Kernel.h"
#define SALT 186
#include "../CUDA10_Registers_Kernel.h"
#define SALT 187
#include "../CUDA10_Registers_Kernel.h"
#define SALT 188
#include "../CUDA10_Registers_Kernel.h"
#define SALT 189
#include "../CUDA10_Registers_Kernel.h"
#define SALT 190
#include "../CUDA10_Registers_Kernel.h"
#define SALT 191
#include "../CUDA10_Registers_Kernel.h"
#define SALT 192
#include "../CUDA10_Registers_Kernel.h"
#define SALT 193
#include "../CUDA10_Registers_Kernel.h"
#define SALT 194
#include "../CUDA10_Registers_Kernel.h"
#define SALT 195
#include "../CUDA10_Registers_Kernel.h"
#define SALT 196
#include "../CUDA10_Registers_Kernel.h"
#define SALT 197
#include "../CUDA10_Registers_Kernel.h"
#define SALT 198
#include "../CUDA10_Registers_Kernel.h"
#define SALT 199
#include "../CUDA10_Registers_Kernel.h"
#define SALT 200
#include "../CUDA10_Registers_Kernel.h"
#define SALT 201
#include "../CUDA10_Registers_Kernel.h"
#define SALT 202
#include "../CUDA10_Registers_Kernel.h"
#define SALT 203
#include "../CUDA10_Registers_Kernel.h"
#define SALT 204
#include "../CUDA10_Registers_Kernel.h"
#define SALT 205
#include "../CUDA10_Registers_Kernel.h"
#define SALT 206
#include "../CUDA10_Registers_Kernel.h"
#define SALT 207
#include "../CUDA10_Registers_Kernel.h"
#define SALT 208
#include "../CUDA10_Registers_Kernel.h"
#define SALT 209
#include "../CUDA10_Registers_Kernel.h"
#define SALT 210
#include "../CUDA10_Registers_Kernel.h"
#define SALT 211
#include "../CUDA10_Registers_Kernel.h"
#define SALT 212
#include "../CUDA10_Registers_Kernel.h"
#define SALT 213
#include "../CUDA10_Registers_Kernel.h"
#define SALT 214
#include "../CUDA10_Registers_Kernel.h"
#define SALT 215
#include "../CUDA10_Registers_Kernel.h"
#define SALT 216
#include "../CUDA10_Registers_Kernel.h"
#define SALT 217
#include "../CUDA10_Registers_Kernel.h"
#define SALT 218
#include "../CUDA10_Registers_Kernel.h"
#define SALT 219
#include "../CUDA10_Registers_Kernel.h"
#define SALT 220
#include "../CUDA10_Registers_Kernel.h"
#define SALT 221
#include "../CUDA10_Registers_Kernel.h"
#define SALT 222
#include "../CUDA10_Registers_Kernel.h"
#define SALT 223
#include "../CUDA10_Registers_Kernel.h"
#define SALT 224
#include "../CUDA10_Registers_Kernel.h"
#define SALT 225
#include "../CUDA10_Registers_Kernel.h"
#define SALT 226
#include "../CUDA10_Registers_Kernel.h"
#define SALT 227
#include "../CUDA10_Registers_Kernel.h"
#define SALT 228
#include "../CUDA10_Registers_Kernel.h"
#define SALT 229
#include "../CUDA10_Registers_Kernel.h"
#define SALT 230
#include "../CUDA10_Registers_Kernel.h"
#define SALT 231
#include "../CUDA10_Registers_Kernel.h"
#define SALT 232
#include "../CUDA10_Registers_Kernel.h"
#define SALT 233
#include "../CUDA10_Registers_Kernel.h"
#define SALT 234
#include "../CUDA10_Registers_Kernel.h"
#define SALT 235
#include "../CUDA10_Registers_Kernel.h"
#define SALT 236
#include "../CUDA10_Registers_Kernel.h"
#define SALT 237
#include "../CUDA10_Registers_Kernel.h"
#define SALT 238
#include "../CUDA10_Registers_Kernel.h"
#define SALT 239
#include "../CUDA10_Registers_Kernel.h"
#define SALT 240
#include "../CUDA10_Registers_Kernel.h"
#define SALT 241
#include "../CUDA10_Registers_Kernel.h"
#define SALT 242
#include "../CUDA10_Registers_Kernel.h"
#define SALT 243
#include "../CUDA10_Registers_Kernel.h"
#define SALT 244
#include "../CUDA10_Registers_Kernel.h"
#define SALT 245
#include "../CUDA10_Registers_Kernel.h"
#define SALT 246
#include "../CUDA10_Registers_Kernel.h"
#define SALT 247
#include "../CUDA10_Registers_Kernel.h"
#define SALT 248
#include "../CUDA10_Registers_Kernel.h"
#define SALT 249
#include "../CUDA10_Registers_Kernel.h"
#define SALT 250
#include "../CUDA10_Registers_Kernel.h"
#define SALT 251
#include "../CUDA10_Registers_Kernel.h"
#define SALT 252
#include "../CUDA10_Registers_Kernel.h"
#define SALT 253
#include "../CUDA10_Registers_Kernel.h"
#define SALT 254
#include "../CUDA10_Registers_Kernel.h"
#define SALT 255
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher0()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel0(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
		case 0: LAUNCH_KERNEL(0); break;
		case 1: LAUNCH_KERNEL(1); break;
		case 2: LAUNCH_KERNEL(2); break;
		case 3: LAUNCH_KERNEL(3); break;
		case 4: LAUNCH_KERNEL(4); break;
		case 5: LAUNCH_KERNEL(5); break;
		case 6: LAUNCH_KERNEL(6); break;
		case 7: LAUNCH_KERNEL(7); break;
		case 8: LAUNCH_KERNEL(8); break;
		case 9: LAUNCH_KERNEL(9); break;
		case 10: LAUNCH_KERNEL(10); break;
		case 11: LAUNCH_KERNEL(11); break;
		case 12: LAUNCH_KERNEL(12); break;
		case 13: LAUNCH_KERNEL(13); break;
		case 14: LAUNCH_KERNEL(14); break;
		case 15: LAUNCH_KERNEL(15); break;
		case 16: LAUNCH_KERNEL(16); break;
		case 17: LAUNCH_KERNEL(17); break;
		case 18: LAUNCH_KERNEL(18); break;
		case 19: LAUNCH_KERNEL(19); break;
		case 20: LAUNCH_KERNEL(20); break;
		case 21: LAUNCH_KERNEL(21); break;
		case 22: LAUNCH_KERNEL(22); break;
		case 23: LAUNCH_KERNEL(23); break;
		case 24: LAUNCH_KERNEL(24); break;
		case 25: LAUNCH_KERNEL(25); break;
		case 26: LAUNCH_KERNEL(26); break;
		case 27: LAUNCH_KERNEL(27); break;
		case 28: LAUNCH_KERNEL(28); break;
		case 29: LAUNCH_KERNEL(29); break;
		case 30: LAUNCH_KERNEL(30); break;
		case 31: LAUNCH_KERNEL(31); break;
		case 32: LAUNCH_KERNEL(32); break;
		case 33: LAUNCH_KERNEL(33); break;
		case 34: LAUNCH_KERNEL(34); break;
		case 35: LAUNCH_KERNEL(35); break;
		case 36: LAUNCH_KERNEL(36); break;
		case 37: LAUNCH_KERNEL(37); break;
		case 38: LAUNCH_KERNEL(38); break;
		case 39: LAUNCH_KERNEL(39); break;
		case 40: LAUNCH_KERNEL(40); break;
		case 41: LAUNCH_KERNEL(41); break;
		case 42: LAUNCH_KERNEL(42); break;
		case 43: LAUNCH_KERNEL(43); break;
		case 44: LAUNCH_KERNEL(44); break;
		case 45: LAUNCH_KERNEL(45); break;
		case 46: LAUNCH_KERNEL(46); break;
		case 47: LAUNCH_KERNEL(47); break;
		case 48: LAUNCH_KERNEL(48); break;
		case 49: LAUNCH_KERNEL(49); break;
		case 50: LAUNCH_KERNEL(50); break;
		case 51: LAUNCH_KERNEL(51); break;
		case 52: LAUNCH_KERNEL(52); break;
		case 53: LAUNCH_KERNEL(53); break;
		case 54: LAUNCH_KERNEL(54); break;
		case 55: LAUNCH_KERNEL(55); break;
		case 56: LAUNCH_KERNEL(56); break;
		case 57: LAUNCH_KERNEL(57); break;
		case 58: LAUNCH_KERNEL(58); break;
		case 59: LAUNCH_KERNEL(59); break;
		case 60: LAUNCH_KERNEL(60); break;
		case 61: LAUNCH_KERNEL(61); break;
		case 62: LAUNCH_KERNEL(62); break;
		case 63: LAUNCH_KERNEL(63); break;
		case 64: LAUNCH_KERNEL(64); break;
		case 65: LAUNCH_KERNEL(65); break;
		case 66: LAUNCH_KERNEL(66); break;
		case 67: LAUNCH_KERNEL(67); break;
		case 68: LAUNCH_KERNEL(68); break;
		case 69: LAUNCH_KERNEL(69); break;
		case 70: LAUNCH_KERNEL(70); break;
		case 71: LAUNCH_KERNEL(71); break;
		case 72: LAUNCH_KERNEL(72); break;
		case 73: LAUNCH_KERNEL(73); break;
		case 74: LAUNCH_KERNEL(74); break;
		case 75: LAUNCH_KERNEL(75); break;
		case 76: LAUNCH_KERNEL(76); break;
		case 77: LAUNCH_KERNEL(77); break;
		case 78: LAUNCH_KERNEL(78); break;
		case 79: LAUNCH_KERNEL(79); break;
		case 80: LAUNCH_KERNEL(80); break;
		case 81: LAUNCH_KERNEL(81); break;
		case 82: LAUNCH_KERNEL(82); break;
		case 83: LAUNCH_KERNEL(83); break;
		case 84: LAUNCH_KERNEL(84); break;
		case 85: LAUNCH_KERNEL(85); break;
		case 86: LAUNCH_KERNEL(86); break;
		case 87: LAUNCH_KERNEL(87); break;
		case 88: LAUNCH_KERNEL(88); break;
		case 89: LAUNCH_KERNEL(89); break;
		case 90: LAUNCH_KERNEL(90); break;
		case 91: LAUNCH_KERNEL(91); break;
		case 92: LAUNCH_KERNEL(92); break;
		case 93: LAUNCH_KERNEL(93); break;
		case 94: LAUNCH_KERNEL(94); break;
		case 95: LAUNCH_KERNEL(95); break;
		case 96: LAUNCH_KERNEL(96); break;
		case 97: LAUNCH_KERNEL(97); break;
		case 98: LAUNCH_KERNEL(98); break;
		case 99: LAUNCH_KERNEL(99); break;
		case 100: LAUNCH_KERNEL(100); break;
		case 101: LAUNCH_KERNEL(101); break;
		case 102: LAUNCH_KERNEL(102); break;
		case 103: LAUNCH_KERNEL(103); break;
		case 104: LAUNCH_KERNEL(104); break;
		case 105: LAUNCH_KERNEL(105); break;
		case 106: LAUNCH_KERNEL(106); break;
		case 107: LAUNCH_KERNEL(107); break;
		case 108: LAUNCH_KERNEL(108); break;
		case 109: LAUNCH_KERNEL(109); break;
		case 110: LAUNCH_KERNEL(110); break;
		case 111: LAUNCH_KERNEL(111); break;
		case 112: LAUNCH_KERNEL(112); break;
		case 113: LAUNCH_KERNEL(113); break;
		case 114: LAUNCH_KERNEL(114); break;
		case 115: LAUNCH_KERNEL(115); break;
		case 116: LAUNCH_KERNEL(116); break;
		case 117: LAUNCH_KERNEL(117); break;
		case 118: LAUNCH_KERNEL(118); break;
		case 119: LAUNCH_KERNEL(119); break;
		case 120: LAUNCH_KERNEL(120); break;
		case 121: LAUNCH_KERNEL(121); break;
		case 122: LAUNCH_KERNEL(122); break;
		case 123: LAUNCH_KERNEL(123); break;
		case 124: LAUNCH_KERNEL(124); break;
		case 125: LAUNCH_KERNEL(125); break;
		case 126: LAUNCH_KERNEL(126); break;
		case 127: LAUNCH_KERNEL(127); break;
		case 128: LAUNCH_KERNEL(128); break;
		case 129: LAUNCH_KERNEL(129); break;
		case 130: LAUNCH_KERNEL(130); break;
		case 131: LAUNCH_KERNEL(131); break;
		case 132: LAUNCH_KERNEL(132); break;
		case 133: LAUNCH_KERNEL(133); break;
		case 134: LAUNCH_KERNEL(134); break;
		case 135: LAUNCH_KERNEL(135); break;
		case 136: LAUNCH_KERNEL(136); break;
		case 137: LAUNCH_KERNEL(137); break;
		case 138: LAUNCH_KERNEL(138); break;
		case 139: LAUNCH_KERNEL(139); break;
		case 140: LAUNCH_KERNEL(140); break;
		case 141: LAUNCH_KERNEL(141); break;
		case 142: LAUNCH_KERNEL(142); break;
		case 143: LAUNCH_KERNEL(143); break;
		case 144: LAUNCH_KERNEL(144); break;
		case 145: LAUNCH_KERNEL(145); break;
		case 146: LAUNCH_KERNEL(146); break;
		case 147: LAUNCH_KERNEL(147); break;
		case 148: LAUNCH_KERNEL(148); break;
		case 149: LAUNCH_KERNEL(149); break;
		case 150: LAUNCH_KERNEL(150); break;
		case 151: LAUNCH_KERNEL(151); break;
		case 152: LAUNCH_KERNEL(152); break;
		case 153: LAUNCH_KERNEL(153); break;
		case 154: LAUNCH_KERNEL(154); break;
		case 155: LAUNCH_KERNEL(155); break;
		case 156: LAUNCH_KERNEL(156); break;
		case 157: LAUNCH_KERNEL(157); break;
		case 158: LAUNCH_KERNEL(158); break;
		case 159: LAUNCH_KERNEL(159); break;
		case 160: LAUNCH_KERNEL(160); break;
		case 161: LAUNCH_KERNEL(161); break;
		case 162: LAUNCH_KERNEL(162); break;
		case 163: LAUNCH_KERNEL(163); break;
		case 164: LAUNCH_KERNEL(164); break;
		case 165: LAUNCH_KERNEL(165); break;
		case 166: LAUNCH_KERNEL(166); break;
		case 167: LAUNCH_KERNEL(167); break;
		case 168: LAUNCH_KERNEL(168); break;
		case 169: LAUNCH_KERNEL(169); break;
		case 170: LAUNCH_KERNEL(170); break;
		case 171: LAUNCH_KERNEL(171); break;
		case 172: LAUNCH_KERNEL(172); break;
		case 173: LAUNCH_KERNEL(173); break;
		case 174: LAUNCH_KERNEL(174); break;
		case 175: LAUNCH_KERNEL(175); break;
		case 176: LAUNCH_KERNEL(176); break;
		case 177: LAUNCH_KERNEL(177); break;
		case 178: LAUNCH_KERNEL(178); break;
		case 179: LAUNCH_KERNEL(179); break;
		case 180: LAUNCH_KERNEL(180); break;
		case 181: LAUNCH_KERNEL(181); break;
		case 182: LAUNCH_KERNEL(182); break;
		case 183: LAUNCH_KERNEL(183); break;
		case 184: LAUNCH_KERNEL(184); break;
		case 185: LAUNCH_KERNEL(185); break;
		case 186: LAUNCH_KERNEL(186); break;
		case 187: LAUNCH_KERNEL(187); break;
		case 188: LAUNCH_KERNEL(188); break;
		case 189: LAUNCH_KERNEL(189); break;
		case 190: LAUNCH_KERNEL(190); break;
		case 191: LAUNCH_KERNEL(191); break;
		case 192: LAUNCH_KERNEL(192); break;
		case 193: LAUNCH_KERNEL(193); break;
		case 194: LAUNCH_KERNEL(194); break;
		case 195: LAUNCH_KERNEL(195); break;
		case 196: LAUNCH_KERNEL(196); break;
		case 197: LAUNCH_KERNEL(197); break;
		case 198: LAUNCH_KERNEL(198); break;
		case 199: LAUNCH_KERNEL(199); break;
		case 200: LAUNCH_KERNEL(200); break;
		case 201: LAUNCH_KERNEL(201); break;
		case 202: LAUNCH_KERNEL(202); break;
		case 203: LAUNCH_KERNEL(203); break;
		case 204: LAUNCH_KERNEL(204); break;
		case 205: LAUNCH_KERNEL(205); break;
		case 206: LAUNCH_KERNEL(206); break;
		case 207: LAUNCH_KERNEL(207); break;
		case 208: LAUNCH_KERNEL(208); break;
		case 209: LAUNCH_KERNEL(209); break;
		case 210: LAUNCH_KERNEL(210); break;
		case 211: LAUNCH_KERNEL(211); break;
		case 212: LAUNCH_KERNEL(212); break;
		case 213: LAUNCH_KERNEL(213); break;
		case 214: LAUNCH_KERNEL(214); break;
		case 215: LAUNCH_KERNEL(215); break;
		case 216: LAUNCH_KERNEL(216); break;
		case 217: LAUNCH_KERNEL(217); break;
		case 218: LAUNCH_KERNEL(218); break;
		case 219: LAUNCH_KERNEL(219); break;
		case 220: LAUNCH_KERNEL(220); break;
		case 221: LAUNCH_KERNEL(221); break;
		case 222: LAUNCH_KERNEL(222); break;
		case 223: LAUNCH_KERNEL(223); break;
		case 224: LAUNCH_KERNEL(224); break;
		case 225: LAUNCH_KERNEL(225); break;
		case 226: LAUNCH_KERNEL(226); break;
		case 227: LAUNCH_KERNEL(227); break;
		case 228: LAUNCH_KERNEL(228); break;
		case 229: LAUNCH_KERNEL(229); break;
		case 230: LAUNCH_KERNEL(230); break;
		case 231: LAUNCH_KERNEL(231); break;
		case 232: LAUNCH_KERNEL(232); break;
		case 233: LAUNCH_KERNEL(233); break;
		case 234: LAUNCH_KERNEL(234); break;
		case 235: LAUNCH_KERNEL(235); break;
		case 236: LAUNCH_KERNEL(236); break;
		case 237: LAUNCH_KERNEL(237); break;
		case 238: LAUNCH_KERNEL(238); break;
		case 239: LAUNCH_KERNEL(239); break;
		case 240: LAUNCH_KERNEL(240); break;
		case 241: LAUNCH_KERNEL(241); break;
		case 242: LAUNCH_KERNEL(242); break;
		case 243: LAUNCH_KERNEL(243); break;
		case 244: LAUNCH_KERNEL(244); break;
		case 245: LAUNCH_KERNEL(245); break;
		case 246: LAUNCH_KERNEL(246); break;
		case 247: LAUNCH_KERNEL(247); break;
		case 248: LAUNCH_KERNEL(248); break;
		case 249: LAUNCH_KERNEL(249); break;
		case 250: LAUNCH_KERNEL(250); break;
		case 251: LAUNCH_KERNEL(251); break;
		case 252: LAUNCH_KERNEL(252); break;
		case 253: LAUNCH_KERNEL(253); break;
		case 254: LAUNCH_KERNEL(254); break;
		case 255: LAUNCH_KERNEL(255); break;
		default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif