// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 3072
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3073
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3074
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3075
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3076
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3077
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3078
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3079
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3080
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3081
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3082
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3083
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3084
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3085
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3086
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3087
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3088
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3089
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3090
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3091
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3092
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3093
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3094
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3095
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3096
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3097
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3098
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3099
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3100
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3101
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3102
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3103
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3104
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3105
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3106
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3107
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3108
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3109
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3110
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3111
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3112
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3113
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3114
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3115
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3116
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3117
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3118
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3119
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3120
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3121
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3122
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3123
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3124
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3125
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3126
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3127
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3128
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3129
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3130
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3131
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3132
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3133
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3134
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3135
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3136
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3137
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3138
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3139
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3140
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3141
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3142
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3143
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3144
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3145
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3146
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3147
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3148
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3149
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3150
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3151
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3152
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3153
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3154
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3155
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3156
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3157
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3158
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3159
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3160
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3161
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3162
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3163
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3164
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3165
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3166
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3167
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3168
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3169
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3170
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3171
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3172
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3173
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3174
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3175
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3176
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3177
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3178
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3179
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3180
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3181
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3182
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3183
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3184
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3185
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3186
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3187
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3188
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3189
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3190
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3191
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3192
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3193
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3194
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3195
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3196
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3197
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3198
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3199
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3200
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3201
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3202
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3203
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3204
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3205
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3206
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3207
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3208
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3209
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3210
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3211
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3212
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3213
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3214
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3215
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3216
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3217
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3218
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3219
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3220
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3221
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3222
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3223
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3224
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3225
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3226
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3227
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3228
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3229
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3230
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3231
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3232
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3233
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3234
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3235
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3236
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3237
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3238
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3239
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3240
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3241
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3242
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3243
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3244
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3245
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3246
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3247
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3248
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3249
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3250
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3251
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3252
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3253
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3254
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3255
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3256
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3257
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3258
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3259
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3260
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3261
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3262
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3263
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3264
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3265
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3266
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3267
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3268
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3269
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3270
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3271
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3272
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3273
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3274
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3275
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3276
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3277
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3278
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3279
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3280
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3281
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3282
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3283
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3284
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3285
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3286
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3287
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3288
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3289
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3290
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3291
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3292
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3293
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3294
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3295
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3296
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3297
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3298
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3299
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3300
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3301
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3302
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3303
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3304
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3305
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3306
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3307
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3308
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3309
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3310
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3311
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3312
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3313
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3314
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3315
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3316
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3317
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3318
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3319
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3320
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3321
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3322
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3323
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3324
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3325
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3326
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3327
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher12()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel12(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 3072: LAUNCH_KERNEL(3072); break;
	case 3073: LAUNCH_KERNEL(3073); break;
	case 3074: LAUNCH_KERNEL(3074); break;
	case 3075: LAUNCH_KERNEL(3075); break;
	case 3076: LAUNCH_KERNEL(3076); break;
	case 3077: LAUNCH_KERNEL(3077); break;
	case 3078: LAUNCH_KERNEL(3078); break;
	case 3079: LAUNCH_KERNEL(3079); break;
	case 3080: LAUNCH_KERNEL(3080); break;
	case 3081: LAUNCH_KERNEL(3081); break;
	case 3082: LAUNCH_KERNEL(3082); break;
	case 3083: LAUNCH_KERNEL(3083); break;
	case 3084: LAUNCH_KERNEL(3084); break;
	case 3085: LAUNCH_KERNEL(3085); break;
	case 3086: LAUNCH_KERNEL(3086); break;
	case 3087: LAUNCH_KERNEL(3087); break;
	case 3088: LAUNCH_KERNEL(3088); break;
	case 3089: LAUNCH_KERNEL(3089); break;
	case 3090: LAUNCH_KERNEL(3090); break;
	case 3091: LAUNCH_KERNEL(3091); break;
	case 3092: LAUNCH_KERNEL(3092); break;
	case 3093: LAUNCH_KERNEL(3093); break;
	case 3094: LAUNCH_KERNEL(3094); break;
	case 3095: LAUNCH_KERNEL(3095); break;
	case 3096: LAUNCH_KERNEL(3096); break;
	case 3097: LAUNCH_KERNEL(3097); break;
	case 3098: LAUNCH_KERNEL(3098); break;
	case 3099: LAUNCH_KERNEL(3099); break;
	case 3100: LAUNCH_KERNEL(3100); break;
	case 3101: LAUNCH_KERNEL(3101); break;
	case 3102: LAUNCH_KERNEL(3102); break;
	case 3103: LAUNCH_KERNEL(3103); break;
	case 3104: LAUNCH_KERNEL(3104); break;
	case 3105: LAUNCH_KERNEL(3105); break;
	case 3106: LAUNCH_KERNEL(3106); break;
	case 3107: LAUNCH_KERNEL(3107); break;
	case 3108: LAUNCH_KERNEL(3108); break;
	case 3109: LAUNCH_KERNEL(3109); break;
	case 3110: LAUNCH_KERNEL(3110); break;
	case 3111: LAUNCH_KERNEL(3111); break;
	case 3112: LAUNCH_KERNEL(3112); break;
	case 3113: LAUNCH_KERNEL(3113); break;
	case 3114: LAUNCH_KERNEL(3114); break;
	case 3115: LAUNCH_KERNEL(3115); break;
	case 3116: LAUNCH_KERNEL(3116); break;
	case 3117: LAUNCH_KERNEL(3117); break;
	case 3118: LAUNCH_KERNEL(3118); break;
	case 3119: LAUNCH_KERNEL(3119); break;
	case 3120: LAUNCH_KERNEL(3120); break;
	case 3121: LAUNCH_KERNEL(3121); break;
	case 3122: LAUNCH_KERNEL(3122); break;
	case 3123: LAUNCH_KERNEL(3123); break;
	case 3124: LAUNCH_KERNEL(3124); break;
	case 3125: LAUNCH_KERNEL(3125); break;
	case 3126: LAUNCH_KERNEL(3126); break;
	case 3127: LAUNCH_KERNEL(3127); break;
	case 3128: LAUNCH_KERNEL(3128); break;
	case 3129: LAUNCH_KERNEL(3129); break;
	case 3130: LAUNCH_KERNEL(3130); break;
	case 3131: LAUNCH_KERNEL(3131); break;
	case 3132: LAUNCH_KERNEL(3132); break;
	case 3133: LAUNCH_KERNEL(3133); break;
	case 3134: LAUNCH_KERNEL(3134); break;
	case 3135: LAUNCH_KERNEL(3135); break;
	case 3136: LAUNCH_KERNEL(3136); break;
	case 3137: LAUNCH_KERNEL(3137); break;
	case 3138: LAUNCH_KERNEL(3138); break;
	case 3139: LAUNCH_KERNEL(3139); break;
	case 3140: LAUNCH_KERNEL(3140); break;
	case 3141: LAUNCH_KERNEL(3141); break;
	case 3142: LAUNCH_KERNEL(3142); break;
	case 3143: LAUNCH_KERNEL(3143); break;
	case 3144: LAUNCH_KERNEL(3144); break;
	case 3145: LAUNCH_KERNEL(3145); break;
	case 3146: LAUNCH_KERNEL(3146); break;
	case 3147: LAUNCH_KERNEL(3147); break;
	case 3148: LAUNCH_KERNEL(3148); break;
	case 3149: LAUNCH_KERNEL(3149); break;
	case 3150: LAUNCH_KERNEL(3150); break;
	case 3151: LAUNCH_KERNEL(3151); break;
	case 3152: LAUNCH_KERNEL(3152); break;
	case 3153: LAUNCH_KERNEL(3153); break;
	case 3154: LAUNCH_KERNEL(3154); break;
	case 3155: LAUNCH_KERNEL(3155); break;
	case 3156: LAUNCH_KERNEL(3156); break;
	case 3157: LAUNCH_KERNEL(3157); break;
	case 3158: LAUNCH_KERNEL(3158); break;
	case 3159: LAUNCH_KERNEL(3159); break;
	case 3160: LAUNCH_KERNEL(3160); break;
	case 3161: LAUNCH_KERNEL(3161); break;
	case 3162: LAUNCH_KERNEL(3162); break;
	case 3163: LAUNCH_KERNEL(3163); break;
	case 3164: LAUNCH_KERNEL(3164); break;
	case 3165: LAUNCH_KERNEL(3165); break;
	case 3166: LAUNCH_KERNEL(3166); break;
	case 3167: LAUNCH_KERNEL(3167); break;
	case 3168: LAUNCH_KERNEL(3168); break;
	case 3169: LAUNCH_KERNEL(3169); break;
	case 3170: LAUNCH_KERNEL(3170); break;
	case 3171: LAUNCH_KERNEL(3171); break;
	case 3172: LAUNCH_KERNEL(3172); break;
	case 3173: LAUNCH_KERNEL(3173); break;
	case 3174: LAUNCH_KERNEL(3174); break;
	case 3175: LAUNCH_KERNEL(3175); break;
	case 3176: LAUNCH_KERNEL(3176); break;
	case 3177: LAUNCH_KERNEL(3177); break;
	case 3178: LAUNCH_KERNEL(3178); break;
	case 3179: LAUNCH_KERNEL(3179); break;
	case 3180: LAUNCH_KERNEL(3180); break;
	case 3181: LAUNCH_KERNEL(3181); break;
	case 3182: LAUNCH_KERNEL(3182); break;
	case 3183: LAUNCH_KERNEL(3183); break;
	case 3184: LAUNCH_KERNEL(3184); break;
	case 3185: LAUNCH_KERNEL(3185); break;
	case 3186: LAUNCH_KERNEL(3186); break;
	case 3187: LAUNCH_KERNEL(3187); break;
	case 3188: LAUNCH_KERNEL(3188); break;
	case 3189: LAUNCH_KERNEL(3189); break;
	case 3190: LAUNCH_KERNEL(3190); break;
	case 3191: LAUNCH_KERNEL(3191); break;
	case 3192: LAUNCH_KERNEL(3192); break;
	case 3193: LAUNCH_KERNEL(3193); break;
	case 3194: LAUNCH_KERNEL(3194); break;
	case 3195: LAUNCH_KERNEL(3195); break;
	case 3196: LAUNCH_KERNEL(3196); break;
	case 3197: LAUNCH_KERNEL(3197); break;
	case 3198: LAUNCH_KERNEL(3198); break;
	case 3199: LAUNCH_KERNEL(3199); break;
	case 3200: LAUNCH_KERNEL(3200); break;
	case 3201: LAUNCH_KERNEL(3201); break;
	case 3202: LAUNCH_KERNEL(3202); break;
	case 3203: LAUNCH_KERNEL(3203); break;
	case 3204: LAUNCH_KERNEL(3204); break;
	case 3205: LAUNCH_KERNEL(3205); break;
	case 3206: LAUNCH_KERNEL(3206); break;
	case 3207: LAUNCH_KERNEL(3207); break;
	case 3208: LAUNCH_KERNEL(3208); break;
	case 3209: LAUNCH_KERNEL(3209); break;
	case 3210: LAUNCH_KERNEL(3210); break;
	case 3211: LAUNCH_KERNEL(3211); break;
	case 3212: LAUNCH_KERNEL(3212); break;
	case 3213: LAUNCH_KERNEL(3213); break;
	case 3214: LAUNCH_KERNEL(3214); break;
	case 3215: LAUNCH_KERNEL(3215); break;
	case 3216: LAUNCH_KERNEL(3216); break;
	case 3217: LAUNCH_KERNEL(3217); break;
	case 3218: LAUNCH_KERNEL(3218); break;
	case 3219: LAUNCH_KERNEL(3219); break;
	case 3220: LAUNCH_KERNEL(3220); break;
	case 3221: LAUNCH_KERNEL(3221); break;
	case 3222: LAUNCH_KERNEL(3222); break;
	case 3223: LAUNCH_KERNEL(3223); break;
	case 3224: LAUNCH_KERNEL(3224); break;
	case 3225: LAUNCH_KERNEL(3225); break;
	case 3226: LAUNCH_KERNEL(3226); break;
	case 3227: LAUNCH_KERNEL(3227); break;
	case 3228: LAUNCH_KERNEL(3228); break;
	case 3229: LAUNCH_KERNEL(3229); break;
	case 3230: LAUNCH_KERNEL(3230); break;
	case 3231: LAUNCH_KERNEL(3231); break;
	case 3232: LAUNCH_KERNEL(3232); break;
	case 3233: LAUNCH_KERNEL(3233); break;
	case 3234: LAUNCH_KERNEL(3234); break;
	case 3235: LAUNCH_KERNEL(3235); break;
	case 3236: LAUNCH_KERNEL(3236); break;
	case 3237: LAUNCH_KERNEL(3237); break;
	case 3238: LAUNCH_KERNEL(3238); break;
	case 3239: LAUNCH_KERNEL(3239); break;
	case 3240: LAUNCH_KERNEL(3240); break;
	case 3241: LAUNCH_KERNEL(3241); break;
	case 3242: LAUNCH_KERNEL(3242); break;
	case 3243: LAUNCH_KERNEL(3243); break;
	case 3244: LAUNCH_KERNEL(3244); break;
	case 3245: LAUNCH_KERNEL(3245); break;
	case 3246: LAUNCH_KERNEL(3246); break;
	case 3247: LAUNCH_KERNEL(3247); break;
	case 3248: LAUNCH_KERNEL(3248); break;
	case 3249: LAUNCH_KERNEL(3249); break;
	case 3250: LAUNCH_KERNEL(3250); break;
	case 3251: LAUNCH_KERNEL(3251); break;
	case 3252: LAUNCH_KERNEL(3252); break;
	case 3253: LAUNCH_KERNEL(3253); break;
	case 3254: LAUNCH_KERNEL(3254); break;
	case 3255: LAUNCH_KERNEL(3255); break;
	case 3256: LAUNCH_KERNEL(3256); break;
	case 3257: LAUNCH_KERNEL(3257); break;
	case 3258: LAUNCH_KERNEL(3258); break;
	case 3259: LAUNCH_KERNEL(3259); break;
	case 3260: LAUNCH_KERNEL(3260); break;
	case 3261: LAUNCH_KERNEL(3261); break;
	case 3262: LAUNCH_KERNEL(3262); break;
	case 3263: LAUNCH_KERNEL(3263); break;
	case 3264: LAUNCH_KERNEL(3264); break;
	case 3265: LAUNCH_KERNEL(3265); break;
	case 3266: LAUNCH_KERNEL(3266); break;
	case 3267: LAUNCH_KERNEL(3267); break;
	case 3268: LAUNCH_KERNEL(3268); break;
	case 3269: LAUNCH_KERNEL(3269); break;
	case 3270: LAUNCH_KERNEL(3270); break;
	case 3271: LAUNCH_KERNEL(3271); break;
	case 3272: LAUNCH_KERNEL(3272); break;
	case 3273: LAUNCH_KERNEL(3273); break;
	case 3274: LAUNCH_KERNEL(3274); break;
	case 3275: LAUNCH_KERNEL(3275); break;
	case 3276: LAUNCH_KERNEL(3276); break;
	case 3277: LAUNCH_KERNEL(3277); break;
	case 3278: LAUNCH_KERNEL(3278); break;
	case 3279: LAUNCH_KERNEL(3279); break;
	case 3280: LAUNCH_KERNEL(3280); break;
	case 3281: LAUNCH_KERNEL(3281); break;
	case 3282: LAUNCH_KERNEL(3282); break;
	case 3283: LAUNCH_KERNEL(3283); break;
	case 3284: LAUNCH_KERNEL(3284); break;
	case 3285: LAUNCH_KERNEL(3285); break;
	case 3286: LAUNCH_KERNEL(3286); break;
	case 3287: LAUNCH_KERNEL(3287); break;
	case 3288: LAUNCH_KERNEL(3288); break;
	case 3289: LAUNCH_KERNEL(3289); break;
	case 3290: LAUNCH_KERNEL(3290); break;
	case 3291: LAUNCH_KERNEL(3291); break;
	case 3292: LAUNCH_KERNEL(3292); break;
	case 3293: LAUNCH_KERNEL(3293); break;
	case 3294: LAUNCH_KERNEL(3294); break;
	case 3295: LAUNCH_KERNEL(3295); break;
	case 3296: LAUNCH_KERNEL(3296); break;
	case 3297: LAUNCH_KERNEL(3297); break;
	case 3298: LAUNCH_KERNEL(3298); break;
	case 3299: LAUNCH_KERNEL(3299); break;
	case 3300: LAUNCH_KERNEL(3300); break;
	case 3301: LAUNCH_KERNEL(3301); break;
	case 3302: LAUNCH_KERNEL(3302); break;
	case 3303: LAUNCH_KERNEL(3303); break;
	case 3304: LAUNCH_KERNEL(3304); break;
	case 3305: LAUNCH_KERNEL(3305); break;
	case 3306: LAUNCH_KERNEL(3306); break;
	case 3307: LAUNCH_KERNEL(3307); break;
	case 3308: LAUNCH_KERNEL(3308); break;
	case 3309: LAUNCH_KERNEL(3309); break;
	case 3310: LAUNCH_KERNEL(3310); break;
	case 3311: LAUNCH_KERNEL(3311); break;
	case 3312: LAUNCH_KERNEL(3312); break;
	case 3313: LAUNCH_KERNEL(3313); break;
	case 3314: LAUNCH_KERNEL(3314); break;
	case 3315: LAUNCH_KERNEL(3315); break;
	case 3316: LAUNCH_KERNEL(3316); break;
	case 3317: LAUNCH_KERNEL(3317); break;
	case 3318: LAUNCH_KERNEL(3318); break;
	case 3319: LAUNCH_KERNEL(3319); break;
	case 3320: LAUNCH_KERNEL(3320); break;
	case 3321: LAUNCH_KERNEL(3321); break;
	case 3322: LAUNCH_KERNEL(3322); break;
	case 3323: LAUNCH_KERNEL(3323); break;
	case 3324: LAUNCH_KERNEL(3324); break;
	case 3325: LAUNCH_KERNEL(3325); break;
	case 3326: LAUNCH_KERNEL(3326); break;
	case 3327: LAUNCH_KERNEL(3327); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
