// Meriken's Tripcode Engine 2.0.1
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 2048
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2049
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2050
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2051
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2052
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2053
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2054
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2055
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2056
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2057
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2058
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2059
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2060
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2061
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2062
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2063
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2064
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2065
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2066
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2067
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2068
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2069
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2070
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2071
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2072
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2073
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2074
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2075
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2076
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2077
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2078
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2079
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2080
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2081
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2082
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2083
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2084
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2085
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2086
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2087
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2088
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2089
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2090
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2091
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2092
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2093
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2094
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2095
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2096
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2097
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2098
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2099
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2100
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2101
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2102
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2103
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2104
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2105
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2106
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2107
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2108
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2109
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2110
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2111
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2112
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2113
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2114
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2115
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2116
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2117
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2118
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2119
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2120
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2121
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2122
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2123
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2124
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2125
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2126
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2127
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2128
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2129
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2130
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2131
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2132
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2133
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2134
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2135
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2136
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2137
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2138
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2139
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2140
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2141
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2142
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2143
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2144
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2145
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2146
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2147
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2148
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2149
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2150
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2151
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2152
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2153
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2154
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2155
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2156
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2157
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2158
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2159
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2160
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2161
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2162
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2163
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2164
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2165
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2166
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2167
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2168
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2169
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2170
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2171
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2172
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2173
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2174
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2175
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2176
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2177
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2178
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2179
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2180
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2181
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2182
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2183
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2184
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2185
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2186
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2187
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2188
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2189
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2190
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2191
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2192
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2193
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2194
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2195
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2196
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2197
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2198
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2199
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2200
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2201
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2202
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2203
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2204
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2205
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2206
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2207
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2208
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2209
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2210
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2211
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2212
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2213
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2214
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2215
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2216
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2217
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2218
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2219
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2220
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2221
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2222
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2223
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2224
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2225
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2226
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2227
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2228
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2229
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2230
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2231
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2232
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2233
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2234
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2235
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2236
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2237
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2238
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2239
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2240
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2241
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2242
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2243
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2244
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2245
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2246
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2247
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2248
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2249
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2250
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2251
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2252
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2253
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2254
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2255
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2256
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2257
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2258
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2259
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2260
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2261
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2262
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2263
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2264
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2265
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2266
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2267
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2268
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2269
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2270
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2271
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2272
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2273
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2274
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2275
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2276
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2277
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2278
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2279
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2280
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2281
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2282
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2283
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2284
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2285
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2286
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2287
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2288
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2289
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2290
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2291
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2292
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2293
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2294
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2295
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2296
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2297
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2298
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2299
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2300
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2301
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2302
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2303
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher8()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel8(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 2048: LAUNCH_KERNEL(2048); break;
	case 2049: LAUNCH_KERNEL(2049); break;
	case 2050: LAUNCH_KERNEL(2050); break;
	case 2051: LAUNCH_KERNEL(2051); break;
	case 2052: LAUNCH_KERNEL(2052); break;
	case 2053: LAUNCH_KERNEL(2053); break;
	case 2054: LAUNCH_KERNEL(2054); break;
	case 2055: LAUNCH_KERNEL(2055); break;
	case 2056: LAUNCH_KERNEL(2056); break;
	case 2057: LAUNCH_KERNEL(2057); break;
	case 2058: LAUNCH_KERNEL(2058); break;
	case 2059: LAUNCH_KERNEL(2059); break;
	case 2060: LAUNCH_KERNEL(2060); break;
	case 2061: LAUNCH_KERNEL(2061); break;
	case 2062: LAUNCH_KERNEL(2062); break;
	case 2063: LAUNCH_KERNEL(2063); break;
	case 2064: LAUNCH_KERNEL(2064); break;
	case 2065: LAUNCH_KERNEL(2065); break;
	case 2066: LAUNCH_KERNEL(2066); break;
	case 2067: LAUNCH_KERNEL(2067); break;
	case 2068: LAUNCH_KERNEL(2068); break;
	case 2069: LAUNCH_KERNEL(2069); break;
	case 2070: LAUNCH_KERNEL(2070); break;
	case 2071: LAUNCH_KERNEL(2071); break;
	case 2072: LAUNCH_KERNEL(2072); break;
	case 2073: LAUNCH_KERNEL(2073); break;
	case 2074: LAUNCH_KERNEL(2074); break;
	case 2075: LAUNCH_KERNEL(2075); break;
	case 2076: LAUNCH_KERNEL(2076); break;
	case 2077: LAUNCH_KERNEL(2077); break;
	case 2078: LAUNCH_KERNEL(2078); break;
	case 2079: LAUNCH_KERNEL(2079); break;
	case 2080: LAUNCH_KERNEL(2080); break;
	case 2081: LAUNCH_KERNEL(2081); break;
	case 2082: LAUNCH_KERNEL(2082); break;
	case 2083: LAUNCH_KERNEL(2083); break;
	case 2084: LAUNCH_KERNEL(2084); break;
	case 2085: LAUNCH_KERNEL(2085); break;
	case 2086: LAUNCH_KERNEL(2086); break;
	case 2087: LAUNCH_KERNEL(2087); break;
	case 2088: LAUNCH_KERNEL(2088); break;
	case 2089: LAUNCH_KERNEL(2089); break;
	case 2090: LAUNCH_KERNEL(2090); break;
	case 2091: LAUNCH_KERNEL(2091); break;
	case 2092: LAUNCH_KERNEL(2092); break;
	case 2093: LAUNCH_KERNEL(2093); break;
	case 2094: LAUNCH_KERNEL(2094); break;
	case 2095: LAUNCH_KERNEL(2095); break;
	case 2096: LAUNCH_KERNEL(2096); break;
	case 2097: LAUNCH_KERNEL(2097); break;
	case 2098: LAUNCH_KERNEL(2098); break;
	case 2099: LAUNCH_KERNEL(2099); break;
	case 2100: LAUNCH_KERNEL(2100); break;
	case 2101: LAUNCH_KERNEL(2101); break;
	case 2102: LAUNCH_KERNEL(2102); break;
	case 2103: LAUNCH_KERNEL(2103); break;
	case 2104: LAUNCH_KERNEL(2104); break;
	case 2105: LAUNCH_KERNEL(2105); break;
	case 2106: LAUNCH_KERNEL(2106); break;
	case 2107: LAUNCH_KERNEL(2107); break;
	case 2108: LAUNCH_KERNEL(2108); break;
	case 2109: LAUNCH_KERNEL(2109); break;
	case 2110: LAUNCH_KERNEL(2110); break;
	case 2111: LAUNCH_KERNEL(2111); break;
	case 2112: LAUNCH_KERNEL(2112); break;
	case 2113: LAUNCH_KERNEL(2113); break;
	case 2114: LAUNCH_KERNEL(2114); break;
	case 2115: LAUNCH_KERNEL(2115); break;
	case 2116: LAUNCH_KERNEL(2116); break;
	case 2117: LAUNCH_KERNEL(2117); break;
	case 2118: LAUNCH_KERNEL(2118); break;
	case 2119: LAUNCH_KERNEL(2119); break;
	case 2120: LAUNCH_KERNEL(2120); break;
	case 2121: LAUNCH_KERNEL(2121); break;
	case 2122: LAUNCH_KERNEL(2122); break;
	case 2123: LAUNCH_KERNEL(2123); break;
	case 2124: LAUNCH_KERNEL(2124); break;
	case 2125: LAUNCH_KERNEL(2125); break;
	case 2126: LAUNCH_KERNEL(2126); break;
	case 2127: LAUNCH_KERNEL(2127); break;
	case 2128: LAUNCH_KERNEL(2128); break;
	case 2129: LAUNCH_KERNEL(2129); break;
	case 2130: LAUNCH_KERNEL(2130); break;
	case 2131: LAUNCH_KERNEL(2131); break;
	case 2132: LAUNCH_KERNEL(2132); break;
	case 2133: LAUNCH_KERNEL(2133); break;
	case 2134: LAUNCH_KERNEL(2134); break;
	case 2135: LAUNCH_KERNEL(2135); break;
	case 2136: LAUNCH_KERNEL(2136); break;
	case 2137: LAUNCH_KERNEL(2137); break;
	case 2138: LAUNCH_KERNEL(2138); break;
	case 2139: LAUNCH_KERNEL(2139); break;
	case 2140: LAUNCH_KERNEL(2140); break;
	case 2141: LAUNCH_KERNEL(2141); break;
	case 2142: LAUNCH_KERNEL(2142); break;
	case 2143: LAUNCH_KERNEL(2143); break;
	case 2144: LAUNCH_KERNEL(2144); break;
	case 2145: LAUNCH_KERNEL(2145); break;
	case 2146: LAUNCH_KERNEL(2146); break;
	case 2147: LAUNCH_KERNEL(2147); break;
	case 2148: LAUNCH_KERNEL(2148); break;
	case 2149: LAUNCH_KERNEL(2149); break;
	case 2150: LAUNCH_KERNEL(2150); break;
	case 2151: LAUNCH_KERNEL(2151); break;
	case 2152: LAUNCH_KERNEL(2152); break;
	case 2153: LAUNCH_KERNEL(2153); break;
	case 2154: LAUNCH_KERNEL(2154); break;
	case 2155: LAUNCH_KERNEL(2155); break;
	case 2156: LAUNCH_KERNEL(2156); break;
	case 2157: LAUNCH_KERNEL(2157); break;
	case 2158: LAUNCH_KERNEL(2158); break;
	case 2159: LAUNCH_KERNEL(2159); break;
	case 2160: LAUNCH_KERNEL(2160); break;
	case 2161: LAUNCH_KERNEL(2161); break;
	case 2162: LAUNCH_KERNEL(2162); break;
	case 2163: LAUNCH_KERNEL(2163); break;
	case 2164: LAUNCH_KERNEL(2164); break;
	case 2165: LAUNCH_KERNEL(2165); break;
	case 2166: LAUNCH_KERNEL(2166); break;
	case 2167: LAUNCH_KERNEL(2167); break;
	case 2168: LAUNCH_KERNEL(2168); break;
	case 2169: LAUNCH_KERNEL(2169); break;
	case 2170: LAUNCH_KERNEL(2170); break;
	case 2171: LAUNCH_KERNEL(2171); break;
	case 2172: LAUNCH_KERNEL(2172); break;
	case 2173: LAUNCH_KERNEL(2173); break;
	case 2174: LAUNCH_KERNEL(2174); break;
	case 2175: LAUNCH_KERNEL(2175); break;
	case 2176: LAUNCH_KERNEL(2176); break;
	case 2177: LAUNCH_KERNEL(2177); break;
	case 2178: LAUNCH_KERNEL(2178); break;
	case 2179: LAUNCH_KERNEL(2179); break;
	case 2180: LAUNCH_KERNEL(2180); break;
	case 2181: LAUNCH_KERNEL(2181); break;
	case 2182: LAUNCH_KERNEL(2182); break;
	case 2183: LAUNCH_KERNEL(2183); break;
	case 2184: LAUNCH_KERNEL(2184); break;
	case 2185: LAUNCH_KERNEL(2185); break;
	case 2186: LAUNCH_KERNEL(2186); break;
	case 2187: LAUNCH_KERNEL(2187); break;
	case 2188: LAUNCH_KERNEL(2188); break;
	case 2189: LAUNCH_KERNEL(2189); break;
	case 2190: LAUNCH_KERNEL(2190); break;
	case 2191: LAUNCH_KERNEL(2191); break;
	case 2192: LAUNCH_KERNEL(2192); break;
	case 2193: LAUNCH_KERNEL(2193); break;
	case 2194: LAUNCH_KERNEL(2194); break;
	case 2195: LAUNCH_KERNEL(2195); break;
	case 2196: LAUNCH_KERNEL(2196); break;
	case 2197: LAUNCH_KERNEL(2197); break;
	case 2198: LAUNCH_KERNEL(2198); break;
	case 2199: LAUNCH_KERNEL(2199); break;
	case 2200: LAUNCH_KERNEL(2200); break;
	case 2201: LAUNCH_KERNEL(2201); break;
	case 2202: LAUNCH_KERNEL(2202); break;
	case 2203: LAUNCH_KERNEL(2203); break;
	case 2204: LAUNCH_KERNEL(2204); break;
	case 2205: LAUNCH_KERNEL(2205); break;
	case 2206: LAUNCH_KERNEL(2206); break;
	case 2207: LAUNCH_KERNEL(2207); break;
	case 2208: LAUNCH_KERNEL(2208); break;
	case 2209: LAUNCH_KERNEL(2209); break;
	case 2210: LAUNCH_KERNEL(2210); break;
	case 2211: LAUNCH_KERNEL(2211); break;
	case 2212: LAUNCH_KERNEL(2212); break;
	case 2213: LAUNCH_KERNEL(2213); break;
	case 2214: LAUNCH_KERNEL(2214); break;
	case 2215: LAUNCH_KERNEL(2215); break;
	case 2216: LAUNCH_KERNEL(2216); break;
	case 2217: LAUNCH_KERNEL(2217); break;
	case 2218: LAUNCH_KERNEL(2218); break;
	case 2219: LAUNCH_KERNEL(2219); break;
	case 2220: LAUNCH_KERNEL(2220); break;
	case 2221: LAUNCH_KERNEL(2221); break;
	case 2222: LAUNCH_KERNEL(2222); break;
	case 2223: LAUNCH_KERNEL(2223); break;
	case 2224: LAUNCH_KERNEL(2224); break;
	case 2225: LAUNCH_KERNEL(2225); break;
	case 2226: LAUNCH_KERNEL(2226); break;
	case 2227: LAUNCH_KERNEL(2227); break;
	case 2228: LAUNCH_KERNEL(2228); break;
	case 2229: LAUNCH_KERNEL(2229); break;
	case 2230: LAUNCH_KERNEL(2230); break;
	case 2231: LAUNCH_KERNEL(2231); break;
	case 2232: LAUNCH_KERNEL(2232); break;
	case 2233: LAUNCH_KERNEL(2233); break;
	case 2234: LAUNCH_KERNEL(2234); break;
	case 2235: LAUNCH_KERNEL(2235); break;
	case 2236: LAUNCH_KERNEL(2236); break;
	case 2237: LAUNCH_KERNEL(2237); break;
	case 2238: LAUNCH_KERNEL(2238); break;
	case 2239: LAUNCH_KERNEL(2239); break;
	case 2240: LAUNCH_KERNEL(2240); break;
	case 2241: LAUNCH_KERNEL(2241); break;
	case 2242: LAUNCH_KERNEL(2242); break;
	case 2243: LAUNCH_KERNEL(2243); break;
	case 2244: LAUNCH_KERNEL(2244); break;
	case 2245: LAUNCH_KERNEL(2245); break;
	case 2246: LAUNCH_KERNEL(2246); break;
	case 2247: LAUNCH_KERNEL(2247); break;
	case 2248: LAUNCH_KERNEL(2248); break;
	case 2249: LAUNCH_KERNEL(2249); break;
	case 2250: LAUNCH_KERNEL(2250); break;
	case 2251: LAUNCH_KERNEL(2251); break;
	case 2252: LAUNCH_KERNEL(2252); break;
	case 2253: LAUNCH_KERNEL(2253); break;
	case 2254: LAUNCH_KERNEL(2254); break;
	case 2255: LAUNCH_KERNEL(2255); break;
	case 2256: LAUNCH_KERNEL(2256); break;
	case 2257: LAUNCH_KERNEL(2257); break;
	case 2258: LAUNCH_KERNEL(2258); break;
	case 2259: LAUNCH_KERNEL(2259); break;
	case 2260: LAUNCH_KERNEL(2260); break;
	case 2261: LAUNCH_KERNEL(2261); break;
	case 2262: LAUNCH_KERNEL(2262); break;
	case 2263: LAUNCH_KERNEL(2263); break;
	case 2264: LAUNCH_KERNEL(2264); break;
	case 2265: LAUNCH_KERNEL(2265); break;
	case 2266: LAUNCH_KERNEL(2266); break;
	case 2267: LAUNCH_KERNEL(2267); break;
	case 2268: LAUNCH_KERNEL(2268); break;
	case 2269: LAUNCH_KERNEL(2269); break;
	case 2270: LAUNCH_KERNEL(2270); break;
	case 2271: LAUNCH_KERNEL(2271); break;
	case 2272: LAUNCH_KERNEL(2272); break;
	case 2273: LAUNCH_KERNEL(2273); break;
	case 2274: LAUNCH_KERNEL(2274); break;
	case 2275: LAUNCH_KERNEL(2275); break;
	case 2276: LAUNCH_KERNEL(2276); break;
	case 2277: LAUNCH_KERNEL(2277); break;
	case 2278: LAUNCH_KERNEL(2278); break;
	case 2279: LAUNCH_KERNEL(2279); break;
	case 2280: LAUNCH_KERNEL(2280); break;
	case 2281: LAUNCH_KERNEL(2281); break;
	case 2282: LAUNCH_KERNEL(2282); break;
	case 2283: LAUNCH_KERNEL(2283); break;
	case 2284: LAUNCH_KERNEL(2284); break;
	case 2285: LAUNCH_KERNEL(2285); break;
	case 2286: LAUNCH_KERNEL(2286); break;
	case 2287: LAUNCH_KERNEL(2287); break;
	case 2288: LAUNCH_KERNEL(2288); break;
	case 2289: LAUNCH_KERNEL(2289); break;
	case 2290: LAUNCH_KERNEL(2290); break;
	case 2291: LAUNCH_KERNEL(2291); break;
	case 2292: LAUNCH_KERNEL(2292); break;
	case 2293: LAUNCH_KERNEL(2293); break;
	case 2294: LAUNCH_KERNEL(2294); break;
	case 2295: LAUNCH_KERNEL(2295); break;
	case 2296: LAUNCH_KERNEL(2296); break;
	case 2297: LAUNCH_KERNEL(2297); break;
	case 2298: LAUNCH_KERNEL(2298); break;
	case 2299: LAUNCH_KERNEL(2299); break;
	case 2300: LAUNCH_KERNEL(2300); break;
	case 2301: LAUNCH_KERNEL(2301); break;
	case 2302: LAUNCH_KERNEL(2302); break;
	case 2303: LAUNCH_KERNEL(2303); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
