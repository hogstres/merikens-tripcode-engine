// Meriken's Tripcode Engine
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 3584
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3585
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3586
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3587
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3588
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3589
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3590
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3591
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3592
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3593
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3594
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3595
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3596
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3597
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3598
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3599
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3600
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3601
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3602
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3603
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3604
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3605
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3606
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3607
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3608
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3609
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3610
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3611
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3612
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3613
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3614
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3615
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3616
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3617
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3618
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3619
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3620
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3621
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3622
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3623
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3624
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3625
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3626
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3627
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3628
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3629
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3630
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3631
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3632
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3633
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3634
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3635
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3636
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3637
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3638
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3639
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3640
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3641
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3642
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3643
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3644
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3645
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3646
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3647
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3648
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3649
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3650
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3651
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3652
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3653
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3654
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3655
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3656
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3657
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3658
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3659
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3660
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3661
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3662
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3663
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3664
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3665
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3666
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3667
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3668
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3669
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3670
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3671
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3672
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3673
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3674
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3675
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3676
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3677
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3678
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3679
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3680
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3681
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3682
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3683
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3684
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3685
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3686
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3687
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3688
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3689
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3690
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3691
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3692
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3693
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3694
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3695
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3696
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3697
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3698
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3699
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3700
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3701
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3702
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3703
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3704
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3705
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3706
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3707
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3708
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3709
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3710
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3711
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3712
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3713
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3714
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3715
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3716
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3717
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3718
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3719
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3720
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3721
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3722
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3723
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3724
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3725
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3726
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3727
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3728
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3729
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3730
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3731
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3732
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3733
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3734
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3735
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3736
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3737
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3738
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3739
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3740
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3741
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3742
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3743
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3744
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3745
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3746
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3747
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3748
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3749
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3750
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3751
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3752
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3753
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3754
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3755
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3756
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3757
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3758
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3759
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3760
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3761
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3762
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3763
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3764
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3765
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3766
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3767
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3768
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3769
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3770
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3771
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3772
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3773
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3774
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3775
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3776
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3777
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3778
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3779
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3780
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3781
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3782
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3783
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3784
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3785
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3786
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3787
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3788
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3789
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3790
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3791
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3792
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3793
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3794
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3795
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3796
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3797
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3798
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3799
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3800
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3801
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3802
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3803
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3804
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3805
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3806
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3807
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3808
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3809
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3810
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3811
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3812
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3813
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3814
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3815
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3816
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3817
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3818
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3819
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3820
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3821
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3822
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3823
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3824
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3825
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3826
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3827
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3828
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3829
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3830
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3831
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3832
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3833
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3834
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3835
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3836
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3837
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3838
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3839
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher14()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel14(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 3584: LAUNCH_KERNEL(3584); break;
	case 3585: LAUNCH_KERNEL(3585); break;
	case 3586: LAUNCH_KERNEL(3586); break;
	case 3587: LAUNCH_KERNEL(3587); break;
	case 3588: LAUNCH_KERNEL(3588); break;
	case 3589: LAUNCH_KERNEL(3589); break;
	case 3590: LAUNCH_KERNEL(3590); break;
	case 3591: LAUNCH_KERNEL(3591); break;
	case 3592: LAUNCH_KERNEL(3592); break;
	case 3593: LAUNCH_KERNEL(3593); break;
	case 3594: LAUNCH_KERNEL(3594); break;
	case 3595: LAUNCH_KERNEL(3595); break;
	case 3596: LAUNCH_KERNEL(3596); break;
	case 3597: LAUNCH_KERNEL(3597); break;
	case 3598: LAUNCH_KERNEL(3598); break;
	case 3599: LAUNCH_KERNEL(3599); break;
	case 3600: LAUNCH_KERNEL(3600); break;
	case 3601: LAUNCH_KERNEL(3601); break;
	case 3602: LAUNCH_KERNEL(3602); break;
	case 3603: LAUNCH_KERNEL(3603); break;
	case 3604: LAUNCH_KERNEL(3604); break;
	case 3605: LAUNCH_KERNEL(3605); break;
	case 3606: LAUNCH_KERNEL(3606); break;
	case 3607: LAUNCH_KERNEL(3607); break;
	case 3608: LAUNCH_KERNEL(3608); break;
	case 3609: LAUNCH_KERNEL(3609); break;
	case 3610: LAUNCH_KERNEL(3610); break;
	case 3611: LAUNCH_KERNEL(3611); break;
	case 3612: LAUNCH_KERNEL(3612); break;
	case 3613: LAUNCH_KERNEL(3613); break;
	case 3614: LAUNCH_KERNEL(3614); break;
	case 3615: LAUNCH_KERNEL(3615); break;
	case 3616: LAUNCH_KERNEL(3616); break;
	case 3617: LAUNCH_KERNEL(3617); break;
	case 3618: LAUNCH_KERNEL(3618); break;
	case 3619: LAUNCH_KERNEL(3619); break;
	case 3620: LAUNCH_KERNEL(3620); break;
	case 3621: LAUNCH_KERNEL(3621); break;
	case 3622: LAUNCH_KERNEL(3622); break;
	case 3623: LAUNCH_KERNEL(3623); break;
	case 3624: LAUNCH_KERNEL(3624); break;
	case 3625: LAUNCH_KERNEL(3625); break;
	case 3626: LAUNCH_KERNEL(3626); break;
	case 3627: LAUNCH_KERNEL(3627); break;
	case 3628: LAUNCH_KERNEL(3628); break;
	case 3629: LAUNCH_KERNEL(3629); break;
	case 3630: LAUNCH_KERNEL(3630); break;
	case 3631: LAUNCH_KERNEL(3631); break;
	case 3632: LAUNCH_KERNEL(3632); break;
	case 3633: LAUNCH_KERNEL(3633); break;
	case 3634: LAUNCH_KERNEL(3634); break;
	case 3635: LAUNCH_KERNEL(3635); break;
	case 3636: LAUNCH_KERNEL(3636); break;
	case 3637: LAUNCH_KERNEL(3637); break;
	case 3638: LAUNCH_KERNEL(3638); break;
	case 3639: LAUNCH_KERNEL(3639); break;
	case 3640: LAUNCH_KERNEL(3640); break;
	case 3641: LAUNCH_KERNEL(3641); break;
	case 3642: LAUNCH_KERNEL(3642); break;
	case 3643: LAUNCH_KERNEL(3643); break;
	case 3644: LAUNCH_KERNEL(3644); break;
	case 3645: LAUNCH_KERNEL(3645); break;
	case 3646: LAUNCH_KERNEL(3646); break;
	case 3647: LAUNCH_KERNEL(3647); break;
	case 3648: LAUNCH_KERNEL(3648); break;
	case 3649: LAUNCH_KERNEL(3649); break;
	case 3650: LAUNCH_KERNEL(3650); break;
	case 3651: LAUNCH_KERNEL(3651); break;
	case 3652: LAUNCH_KERNEL(3652); break;
	case 3653: LAUNCH_KERNEL(3653); break;
	case 3654: LAUNCH_KERNEL(3654); break;
	case 3655: LAUNCH_KERNEL(3655); break;
	case 3656: LAUNCH_KERNEL(3656); break;
	case 3657: LAUNCH_KERNEL(3657); break;
	case 3658: LAUNCH_KERNEL(3658); break;
	case 3659: LAUNCH_KERNEL(3659); break;
	case 3660: LAUNCH_KERNEL(3660); break;
	case 3661: LAUNCH_KERNEL(3661); break;
	case 3662: LAUNCH_KERNEL(3662); break;
	case 3663: LAUNCH_KERNEL(3663); break;
	case 3664: LAUNCH_KERNEL(3664); break;
	case 3665: LAUNCH_KERNEL(3665); break;
	case 3666: LAUNCH_KERNEL(3666); break;
	case 3667: LAUNCH_KERNEL(3667); break;
	case 3668: LAUNCH_KERNEL(3668); break;
	case 3669: LAUNCH_KERNEL(3669); break;
	case 3670: LAUNCH_KERNEL(3670); break;
	case 3671: LAUNCH_KERNEL(3671); break;
	case 3672: LAUNCH_KERNEL(3672); break;
	case 3673: LAUNCH_KERNEL(3673); break;
	case 3674: LAUNCH_KERNEL(3674); break;
	case 3675: LAUNCH_KERNEL(3675); break;
	case 3676: LAUNCH_KERNEL(3676); break;
	case 3677: LAUNCH_KERNEL(3677); break;
	case 3678: LAUNCH_KERNEL(3678); break;
	case 3679: LAUNCH_KERNEL(3679); break;
	case 3680: LAUNCH_KERNEL(3680); break;
	case 3681: LAUNCH_KERNEL(3681); break;
	case 3682: LAUNCH_KERNEL(3682); break;
	case 3683: LAUNCH_KERNEL(3683); break;
	case 3684: LAUNCH_KERNEL(3684); break;
	case 3685: LAUNCH_KERNEL(3685); break;
	case 3686: LAUNCH_KERNEL(3686); break;
	case 3687: LAUNCH_KERNEL(3687); break;
	case 3688: LAUNCH_KERNEL(3688); break;
	case 3689: LAUNCH_KERNEL(3689); break;
	case 3690: LAUNCH_KERNEL(3690); break;
	case 3691: LAUNCH_KERNEL(3691); break;
	case 3692: LAUNCH_KERNEL(3692); break;
	case 3693: LAUNCH_KERNEL(3693); break;
	case 3694: LAUNCH_KERNEL(3694); break;
	case 3695: LAUNCH_KERNEL(3695); break;
	case 3696: LAUNCH_KERNEL(3696); break;
	case 3697: LAUNCH_KERNEL(3697); break;
	case 3698: LAUNCH_KERNEL(3698); break;
	case 3699: LAUNCH_KERNEL(3699); break;
	case 3700: LAUNCH_KERNEL(3700); break;
	case 3701: LAUNCH_KERNEL(3701); break;
	case 3702: LAUNCH_KERNEL(3702); break;
	case 3703: LAUNCH_KERNEL(3703); break;
	case 3704: LAUNCH_KERNEL(3704); break;
	case 3705: LAUNCH_KERNEL(3705); break;
	case 3706: LAUNCH_KERNEL(3706); break;
	case 3707: LAUNCH_KERNEL(3707); break;
	case 3708: LAUNCH_KERNEL(3708); break;
	case 3709: LAUNCH_KERNEL(3709); break;
	case 3710: LAUNCH_KERNEL(3710); break;
	case 3711: LAUNCH_KERNEL(3711); break;
	case 3712: LAUNCH_KERNEL(3712); break;
	case 3713: LAUNCH_KERNEL(3713); break;
	case 3714: LAUNCH_KERNEL(3714); break;
	case 3715: LAUNCH_KERNEL(3715); break;
	case 3716: LAUNCH_KERNEL(3716); break;
	case 3717: LAUNCH_KERNEL(3717); break;
	case 3718: LAUNCH_KERNEL(3718); break;
	case 3719: LAUNCH_KERNEL(3719); break;
	case 3720: LAUNCH_KERNEL(3720); break;
	case 3721: LAUNCH_KERNEL(3721); break;
	case 3722: LAUNCH_KERNEL(3722); break;
	case 3723: LAUNCH_KERNEL(3723); break;
	case 3724: LAUNCH_KERNEL(3724); break;
	case 3725: LAUNCH_KERNEL(3725); break;
	case 3726: LAUNCH_KERNEL(3726); break;
	case 3727: LAUNCH_KERNEL(3727); break;
	case 3728: LAUNCH_KERNEL(3728); break;
	case 3729: LAUNCH_KERNEL(3729); break;
	case 3730: LAUNCH_KERNEL(3730); break;
	case 3731: LAUNCH_KERNEL(3731); break;
	case 3732: LAUNCH_KERNEL(3732); break;
	case 3733: LAUNCH_KERNEL(3733); break;
	case 3734: LAUNCH_KERNEL(3734); break;
	case 3735: LAUNCH_KERNEL(3735); break;
	case 3736: LAUNCH_KERNEL(3736); break;
	case 3737: LAUNCH_KERNEL(3737); break;
	case 3738: LAUNCH_KERNEL(3738); break;
	case 3739: LAUNCH_KERNEL(3739); break;
	case 3740: LAUNCH_KERNEL(3740); break;
	case 3741: LAUNCH_KERNEL(3741); break;
	case 3742: LAUNCH_KERNEL(3742); break;
	case 3743: LAUNCH_KERNEL(3743); break;
	case 3744: LAUNCH_KERNEL(3744); break;
	case 3745: LAUNCH_KERNEL(3745); break;
	case 3746: LAUNCH_KERNEL(3746); break;
	case 3747: LAUNCH_KERNEL(3747); break;
	case 3748: LAUNCH_KERNEL(3748); break;
	case 3749: LAUNCH_KERNEL(3749); break;
	case 3750: LAUNCH_KERNEL(3750); break;
	case 3751: LAUNCH_KERNEL(3751); break;
	case 3752: LAUNCH_KERNEL(3752); break;
	case 3753: LAUNCH_KERNEL(3753); break;
	case 3754: LAUNCH_KERNEL(3754); break;
	case 3755: LAUNCH_KERNEL(3755); break;
	case 3756: LAUNCH_KERNEL(3756); break;
	case 3757: LAUNCH_KERNEL(3757); break;
	case 3758: LAUNCH_KERNEL(3758); break;
	case 3759: LAUNCH_KERNEL(3759); break;
	case 3760: LAUNCH_KERNEL(3760); break;
	case 3761: LAUNCH_KERNEL(3761); break;
	case 3762: LAUNCH_KERNEL(3762); break;
	case 3763: LAUNCH_KERNEL(3763); break;
	case 3764: LAUNCH_KERNEL(3764); break;
	case 3765: LAUNCH_KERNEL(3765); break;
	case 3766: LAUNCH_KERNEL(3766); break;
	case 3767: LAUNCH_KERNEL(3767); break;
	case 3768: LAUNCH_KERNEL(3768); break;
	case 3769: LAUNCH_KERNEL(3769); break;
	case 3770: LAUNCH_KERNEL(3770); break;
	case 3771: LAUNCH_KERNEL(3771); break;
	case 3772: LAUNCH_KERNEL(3772); break;
	case 3773: LAUNCH_KERNEL(3773); break;
	case 3774: LAUNCH_KERNEL(3774); break;
	case 3775: LAUNCH_KERNEL(3775); break;
	case 3776: LAUNCH_KERNEL(3776); break;
	case 3777: LAUNCH_KERNEL(3777); break;
	case 3778: LAUNCH_KERNEL(3778); break;
	case 3779: LAUNCH_KERNEL(3779); break;
	case 3780: LAUNCH_KERNEL(3780); break;
	case 3781: LAUNCH_KERNEL(3781); break;
	case 3782: LAUNCH_KERNEL(3782); break;
	case 3783: LAUNCH_KERNEL(3783); break;
	case 3784: LAUNCH_KERNEL(3784); break;
	case 3785: LAUNCH_KERNEL(3785); break;
	case 3786: LAUNCH_KERNEL(3786); break;
	case 3787: LAUNCH_KERNEL(3787); break;
	case 3788: LAUNCH_KERNEL(3788); break;
	case 3789: LAUNCH_KERNEL(3789); break;
	case 3790: LAUNCH_KERNEL(3790); break;
	case 3791: LAUNCH_KERNEL(3791); break;
	case 3792: LAUNCH_KERNEL(3792); break;
	case 3793: LAUNCH_KERNEL(3793); break;
	case 3794: LAUNCH_KERNEL(3794); break;
	case 3795: LAUNCH_KERNEL(3795); break;
	case 3796: LAUNCH_KERNEL(3796); break;
	case 3797: LAUNCH_KERNEL(3797); break;
	case 3798: LAUNCH_KERNEL(3798); break;
	case 3799: LAUNCH_KERNEL(3799); break;
	case 3800: LAUNCH_KERNEL(3800); break;
	case 3801: LAUNCH_KERNEL(3801); break;
	case 3802: LAUNCH_KERNEL(3802); break;
	case 3803: LAUNCH_KERNEL(3803); break;
	case 3804: LAUNCH_KERNEL(3804); break;
	case 3805: LAUNCH_KERNEL(3805); break;
	case 3806: LAUNCH_KERNEL(3806); break;
	case 3807: LAUNCH_KERNEL(3807); break;
	case 3808: LAUNCH_KERNEL(3808); break;
	case 3809: LAUNCH_KERNEL(3809); break;
	case 3810: LAUNCH_KERNEL(3810); break;
	case 3811: LAUNCH_KERNEL(3811); break;
	case 3812: LAUNCH_KERNEL(3812); break;
	case 3813: LAUNCH_KERNEL(3813); break;
	case 3814: LAUNCH_KERNEL(3814); break;
	case 3815: LAUNCH_KERNEL(3815); break;
	case 3816: LAUNCH_KERNEL(3816); break;
	case 3817: LAUNCH_KERNEL(3817); break;
	case 3818: LAUNCH_KERNEL(3818); break;
	case 3819: LAUNCH_KERNEL(3819); break;
	case 3820: LAUNCH_KERNEL(3820); break;
	case 3821: LAUNCH_KERNEL(3821); break;
	case 3822: LAUNCH_KERNEL(3822); break;
	case 3823: LAUNCH_KERNEL(3823); break;
	case 3824: LAUNCH_KERNEL(3824); break;
	case 3825: LAUNCH_KERNEL(3825); break;
	case 3826: LAUNCH_KERNEL(3826); break;
	case 3827: LAUNCH_KERNEL(3827); break;
	case 3828: LAUNCH_KERNEL(3828); break;
	case 3829: LAUNCH_KERNEL(3829); break;
	case 3830: LAUNCH_KERNEL(3830); break;
	case 3831: LAUNCH_KERNEL(3831); break;
	case 3832: LAUNCH_KERNEL(3832); break;
	case 3833: LAUNCH_KERNEL(3833); break;
	case 3834: LAUNCH_KERNEL(3834); break;
	case 3835: LAUNCH_KERNEL(3835); break;
	case 3836: LAUNCH_KERNEL(3836); break;
	case 3837: LAUNCH_KERNEL(3837); break;
	case 3838: LAUNCH_KERNEL(3838); break;
	case 3839: LAUNCH_KERNEL(3839); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
