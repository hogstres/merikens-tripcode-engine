// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 768
#include "../CUDA10_Registers_Kernel.h"
#define SALT 769
#include "../CUDA10_Registers_Kernel.h"
#define SALT 770
#include "../CUDA10_Registers_Kernel.h"
#define SALT 771
#include "../CUDA10_Registers_Kernel.h"
#define SALT 772
#include "../CUDA10_Registers_Kernel.h"
#define SALT 773
#include "../CUDA10_Registers_Kernel.h"
#define SALT 774
#include "../CUDA10_Registers_Kernel.h"
#define SALT 775
#include "../CUDA10_Registers_Kernel.h"
#define SALT 776
#include "../CUDA10_Registers_Kernel.h"
#define SALT 777
#include "../CUDA10_Registers_Kernel.h"
#define SALT 778
#include "../CUDA10_Registers_Kernel.h"
#define SALT 779
#include "../CUDA10_Registers_Kernel.h"
#define SALT 780
#include "../CUDA10_Registers_Kernel.h"
#define SALT 781
#include "../CUDA10_Registers_Kernel.h"
#define SALT 782
#include "../CUDA10_Registers_Kernel.h"
#define SALT 783
#include "../CUDA10_Registers_Kernel.h"
#define SALT 784
#include "../CUDA10_Registers_Kernel.h"
#define SALT 785
#include "../CUDA10_Registers_Kernel.h"
#define SALT 786
#include "../CUDA10_Registers_Kernel.h"
#define SALT 787
#include "../CUDA10_Registers_Kernel.h"
#define SALT 788
#include "../CUDA10_Registers_Kernel.h"
#define SALT 789
#include "../CUDA10_Registers_Kernel.h"
#define SALT 790
#include "../CUDA10_Registers_Kernel.h"
#define SALT 791
#include "../CUDA10_Registers_Kernel.h"
#define SALT 792
#include "../CUDA10_Registers_Kernel.h"
#define SALT 793
#include "../CUDA10_Registers_Kernel.h"
#define SALT 794
#include "../CUDA10_Registers_Kernel.h"
#define SALT 795
#include "../CUDA10_Registers_Kernel.h"
#define SALT 796
#include "../CUDA10_Registers_Kernel.h"
#define SALT 797
#include "../CUDA10_Registers_Kernel.h"
#define SALT 798
#include "../CUDA10_Registers_Kernel.h"
#define SALT 799
#include "../CUDA10_Registers_Kernel.h"
#define SALT 800
#include "../CUDA10_Registers_Kernel.h"
#define SALT 801
#include "../CUDA10_Registers_Kernel.h"
#define SALT 802
#include "../CUDA10_Registers_Kernel.h"
#define SALT 803
#include "../CUDA10_Registers_Kernel.h"
#define SALT 804
#include "../CUDA10_Registers_Kernel.h"
#define SALT 805
#include "../CUDA10_Registers_Kernel.h"
#define SALT 806
#include "../CUDA10_Registers_Kernel.h"
#define SALT 807
#include "../CUDA10_Registers_Kernel.h"
#define SALT 808
#include "../CUDA10_Registers_Kernel.h"
#define SALT 809
#include "../CUDA10_Registers_Kernel.h"
#define SALT 810
#include "../CUDA10_Registers_Kernel.h"
#define SALT 811
#include "../CUDA10_Registers_Kernel.h"
#define SALT 812
#include "../CUDA10_Registers_Kernel.h"
#define SALT 813
#include "../CUDA10_Registers_Kernel.h"
#define SALT 814
#include "../CUDA10_Registers_Kernel.h"
#define SALT 815
#include "../CUDA10_Registers_Kernel.h"
#define SALT 816
#include "../CUDA10_Registers_Kernel.h"
#define SALT 817
#include "../CUDA10_Registers_Kernel.h"
#define SALT 818
#include "../CUDA10_Registers_Kernel.h"
#define SALT 819
#include "../CUDA10_Registers_Kernel.h"
#define SALT 820
#include "../CUDA10_Registers_Kernel.h"
#define SALT 821
#include "../CUDA10_Registers_Kernel.h"
#define SALT 822
#include "../CUDA10_Registers_Kernel.h"
#define SALT 823
#include "../CUDA10_Registers_Kernel.h"
#define SALT 824
#include "../CUDA10_Registers_Kernel.h"
#define SALT 825
#include "../CUDA10_Registers_Kernel.h"
#define SALT 826
#include "../CUDA10_Registers_Kernel.h"
#define SALT 827
#include "../CUDA10_Registers_Kernel.h"
#define SALT 828
#include "../CUDA10_Registers_Kernel.h"
#define SALT 829
#include "../CUDA10_Registers_Kernel.h"
#define SALT 830
#include "../CUDA10_Registers_Kernel.h"
#define SALT 831
#include "../CUDA10_Registers_Kernel.h"
#define SALT 832
#include "../CUDA10_Registers_Kernel.h"
#define SALT 833
#include "../CUDA10_Registers_Kernel.h"
#define SALT 834
#include "../CUDA10_Registers_Kernel.h"
#define SALT 835
#include "../CUDA10_Registers_Kernel.h"
#define SALT 836
#include "../CUDA10_Registers_Kernel.h"
#define SALT 837
#include "../CUDA10_Registers_Kernel.h"
#define SALT 838
#include "../CUDA10_Registers_Kernel.h"
#define SALT 839
#include "../CUDA10_Registers_Kernel.h"
#define SALT 840
#include "../CUDA10_Registers_Kernel.h"
#define SALT 841
#include "../CUDA10_Registers_Kernel.h"
#define SALT 842
#include "../CUDA10_Registers_Kernel.h"
#define SALT 843
#include "../CUDA10_Registers_Kernel.h"
#define SALT 844
#include "../CUDA10_Registers_Kernel.h"
#define SALT 845
#include "../CUDA10_Registers_Kernel.h"
#define SALT 846
#include "../CUDA10_Registers_Kernel.h"
#define SALT 847
#include "../CUDA10_Registers_Kernel.h"
#define SALT 848
#include "../CUDA10_Registers_Kernel.h"
#define SALT 849
#include "../CUDA10_Registers_Kernel.h"
#define SALT 850
#include "../CUDA10_Registers_Kernel.h"
#define SALT 851
#include "../CUDA10_Registers_Kernel.h"
#define SALT 852
#include "../CUDA10_Registers_Kernel.h"
#define SALT 853
#include "../CUDA10_Registers_Kernel.h"
#define SALT 854
#include "../CUDA10_Registers_Kernel.h"
#define SALT 855
#include "../CUDA10_Registers_Kernel.h"
#define SALT 856
#include "../CUDA10_Registers_Kernel.h"
#define SALT 857
#include "../CUDA10_Registers_Kernel.h"
#define SALT 858
#include "../CUDA10_Registers_Kernel.h"
#define SALT 859
#include "../CUDA10_Registers_Kernel.h"
#define SALT 860
#include "../CUDA10_Registers_Kernel.h"
#define SALT 861
#include "../CUDA10_Registers_Kernel.h"
#define SALT 862
#include "../CUDA10_Registers_Kernel.h"
#define SALT 863
#include "../CUDA10_Registers_Kernel.h"
#define SALT 864
#include "../CUDA10_Registers_Kernel.h"
#define SALT 865
#include "../CUDA10_Registers_Kernel.h"
#define SALT 866
#include "../CUDA10_Registers_Kernel.h"
#define SALT 867
#include "../CUDA10_Registers_Kernel.h"
#define SALT 868
#include "../CUDA10_Registers_Kernel.h"
#define SALT 869
#include "../CUDA10_Registers_Kernel.h"
#define SALT 870
#include "../CUDA10_Registers_Kernel.h"
#define SALT 871
#include "../CUDA10_Registers_Kernel.h"
#define SALT 872
#include "../CUDA10_Registers_Kernel.h"
#define SALT 873
#include "../CUDA10_Registers_Kernel.h"
#define SALT 874
#include "../CUDA10_Registers_Kernel.h"
#define SALT 875
#include "../CUDA10_Registers_Kernel.h"
#define SALT 876
#include "../CUDA10_Registers_Kernel.h"
#define SALT 877
#include "../CUDA10_Registers_Kernel.h"
#define SALT 878
#include "../CUDA10_Registers_Kernel.h"
#define SALT 879
#include "../CUDA10_Registers_Kernel.h"
#define SALT 880
#include "../CUDA10_Registers_Kernel.h"
#define SALT 881
#include "../CUDA10_Registers_Kernel.h"
#define SALT 882
#include "../CUDA10_Registers_Kernel.h"
#define SALT 883
#include "../CUDA10_Registers_Kernel.h"
#define SALT 884
#include "../CUDA10_Registers_Kernel.h"
#define SALT 885
#include "../CUDA10_Registers_Kernel.h"
#define SALT 886
#include "../CUDA10_Registers_Kernel.h"
#define SALT 887
#include "../CUDA10_Registers_Kernel.h"
#define SALT 888
#include "../CUDA10_Registers_Kernel.h"
#define SALT 889
#include "../CUDA10_Registers_Kernel.h"
#define SALT 890
#include "../CUDA10_Registers_Kernel.h"
#define SALT 891
#include "../CUDA10_Registers_Kernel.h"
#define SALT 892
#include "../CUDA10_Registers_Kernel.h"
#define SALT 893
#include "../CUDA10_Registers_Kernel.h"
#define SALT 894
#include "../CUDA10_Registers_Kernel.h"
#define SALT 895
#include "../CUDA10_Registers_Kernel.h"
#define SALT 896
#include "../CUDA10_Registers_Kernel.h"
#define SALT 897
#include "../CUDA10_Registers_Kernel.h"
#define SALT 898
#include "../CUDA10_Registers_Kernel.h"
#define SALT 899
#include "../CUDA10_Registers_Kernel.h"
#define SALT 900
#include "../CUDA10_Registers_Kernel.h"
#define SALT 901
#include "../CUDA10_Registers_Kernel.h"
#define SALT 902
#include "../CUDA10_Registers_Kernel.h"
#define SALT 903
#include "../CUDA10_Registers_Kernel.h"
#define SALT 904
#include "../CUDA10_Registers_Kernel.h"
#define SALT 905
#include "../CUDA10_Registers_Kernel.h"
#define SALT 906
#include "../CUDA10_Registers_Kernel.h"
#define SALT 907
#include "../CUDA10_Registers_Kernel.h"
#define SALT 908
#include "../CUDA10_Registers_Kernel.h"
#define SALT 909
#include "../CUDA10_Registers_Kernel.h"
#define SALT 910
#include "../CUDA10_Registers_Kernel.h"
#define SALT 911
#include "../CUDA10_Registers_Kernel.h"
#define SALT 912
#include "../CUDA10_Registers_Kernel.h"
#define SALT 913
#include "../CUDA10_Registers_Kernel.h"
#define SALT 914
#include "../CUDA10_Registers_Kernel.h"
#define SALT 915
#include "../CUDA10_Registers_Kernel.h"
#define SALT 916
#include "../CUDA10_Registers_Kernel.h"
#define SALT 917
#include "../CUDA10_Registers_Kernel.h"
#define SALT 918
#include "../CUDA10_Registers_Kernel.h"
#define SALT 919
#include "../CUDA10_Registers_Kernel.h"
#define SALT 920
#include "../CUDA10_Registers_Kernel.h"
#define SALT 921
#include "../CUDA10_Registers_Kernel.h"
#define SALT 922
#include "../CUDA10_Registers_Kernel.h"
#define SALT 923
#include "../CUDA10_Registers_Kernel.h"
#define SALT 924
#include "../CUDA10_Registers_Kernel.h"
#define SALT 925
#include "../CUDA10_Registers_Kernel.h"
#define SALT 926
#include "../CUDA10_Registers_Kernel.h"
#define SALT 927
#include "../CUDA10_Registers_Kernel.h"
#define SALT 928
#include "../CUDA10_Registers_Kernel.h"
#define SALT 929
#include "../CUDA10_Registers_Kernel.h"
#define SALT 930
#include "../CUDA10_Registers_Kernel.h"
#define SALT 931
#include "../CUDA10_Registers_Kernel.h"
#define SALT 932
#include "../CUDA10_Registers_Kernel.h"
#define SALT 933
#include "../CUDA10_Registers_Kernel.h"
#define SALT 934
#include "../CUDA10_Registers_Kernel.h"
#define SALT 935
#include "../CUDA10_Registers_Kernel.h"
#define SALT 936
#include "../CUDA10_Registers_Kernel.h"
#define SALT 937
#include "../CUDA10_Registers_Kernel.h"
#define SALT 938
#include "../CUDA10_Registers_Kernel.h"
#define SALT 939
#include "../CUDA10_Registers_Kernel.h"
#define SALT 940
#include "../CUDA10_Registers_Kernel.h"
#define SALT 941
#include "../CUDA10_Registers_Kernel.h"
#define SALT 942
#include "../CUDA10_Registers_Kernel.h"
#define SALT 943
#include "../CUDA10_Registers_Kernel.h"
#define SALT 944
#include "../CUDA10_Registers_Kernel.h"
#define SALT 945
#include "../CUDA10_Registers_Kernel.h"
#define SALT 946
#include "../CUDA10_Registers_Kernel.h"
#define SALT 947
#include "../CUDA10_Registers_Kernel.h"
#define SALT 948
#include "../CUDA10_Registers_Kernel.h"
#define SALT 949
#include "../CUDA10_Registers_Kernel.h"
#define SALT 950
#include "../CUDA10_Registers_Kernel.h"
#define SALT 951
#include "../CUDA10_Registers_Kernel.h"
#define SALT 952
#include "../CUDA10_Registers_Kernel.h"
#define SALT 953
#include "../CUDA10_Registers_Kernel.h"
#define SALT 954
#include "../CUDA10_Registers_Kernel.h"
#define SALT 955
#include "../CUDA10_Registers_Kernel.h"
#define SALT 956
#include "../CUDA10_Registers_Kernel.h"
#define SALT 957
#include "../CUDA10_Registers_Kernel.h"
#define SALT 958
#include "../CUDA10_Registers_Kernel.h"
#define SALT 959
#include "../CUDA10_Registers_Kernel.h"
#define SALT 960
#include "../CUDA10_Registers_Kernel.h"
#define SALT 961
#include "../CUDA10_Registers_Kernel.h"
#define SALT 962
#include "../CUDA10_Registers_Kernel.h"
#define SALT 963
#include "../CUDA10_Registers_Kernel.h"
#define SALT 964
#include "../CUDA10_Registers_Kernel.h"
#define SALT 965
#include "../CUDA10_Registers_Kernel.h"
#define SALT 966
#include "../CUDA10_Registers_Kernel.h"
#define SALT 967
#include "../CUDA10_Registers_Kernel.h"
#define SALT 968
#include "../CUDA10_Registers_Kernel.h"
#define SALT 969
#include "../CUDA10_Registers_Kernel.h"
#define SALT 970
#include "../CUDA10_Registers_Kernel.h"
#define SALT 971
#include "../CUDA10_Registers_Kernel.h"
#define SALT 972
#include "../CUDA10_Registers_Kernel.h"
#define SALT 973
#include "../CUDA10_Registers_Kernel.h"
#define SALT 974
#include "../CUDA10_Registers_Kernel.h"
#define SALT 975
#include "../CUDA10_Registers_Kernel.h"
#define SALT 976
#include "../CUDA10_Registers_Kernel.h"
#define SALT 977
#include "../CUDA10_Registers_Kernel.h"
#define SALT 978
#include "../CUDA10_Registers_Kernel.h"
#define SALT 979
#include "../CUDA10_Registers_Kernel.h"
#define SALT 980
#include "../CUDA10_Registers_Kernel.h"
#define SALT 981
#include "../CUDA10_Registers_Kernel.h"
#define SALT 982
#include "../CUDA10_Registers_Kernel.h"
#define SALT 983
#include "../CUDA10_Registers_Kernel.h"
#define SALT 984
#include "../CUDA10_Registers_Kernel.h"
#define SALT 985
#include "../CUDA10_Registers_Kernel.h"
#define SALT 986
#include "../CUDA10_Registers_Kernel.h"
#define SALT 987
#include "../CUDA10_Registers_Kernel.h"
#define SALT 988
#include "../CUDA10_Registers_Kernel.h"
#define SALT 989
#include "../CUDA10_Registers_Kernel.h"
#define SALT 990
#include "../CUDA10_Registers_Kernel.h"
#define SALT 991
#include "../CUDA10_Registers_Kernel.h"
#define SALT 992
#include "../CUDA10_Registers_Kernel.h"
#define SALT 993
#include "../CUDA10_Registers_Kernel.h"
#define SALT 994
#include "../CUDA10_Registers_Kernel.h"
#define SALT 995
#include "../CUDA10_Registers_Kernel.h"
#define SALT 996
#include "../CUDA10_Registers_Kernel.h"
#define SALT 997
#include "../CUDA10_Registers_Kernel.h"
#define SALT 998
#include "../CUDA10_Registers_Kernel.h"
#define SALT 999
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1000
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1001
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1002
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1003
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1004
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1005
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1006
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1007
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1008
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1009
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1010
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1011
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1012
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1013
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1014
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1015
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1016
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1017
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1018
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1019
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1020
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1021
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1022
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1023
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher3()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel3(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
		case 768: LAUNCH_KERNEL(768); break;
		case 769: LAUNCH_KERNEL(769); break;
		case 770: LAUNCH_KERNEL(770); break;
		case 771: LAUNCH_KERNEL(771); break;
		case 772: LAUNCH_KERNEL(772); break;
		case 773: LAUNCH_KERNEL(773); break;
		case 774: LAUNCH_KERNEL(774); break;
		case 775: LAUNCH_KERNEL(775); break;
		case 776: LAUNCH_KERNEL(776); break;
		case 777: LAUNCH_KERNEL(777); break;
		case 778: LAUNCH_KERNEL(778); break;
		case 779: LAUNCH_KERNEL(779); break;
		case 780: LAUNCH_KERNEL(780); break;
		case 781: LAUNCH_KERNEL(781); break;
		case 782: LAUNCH_KERNEL(782); break;
		case 783: LAUNCH_KERNEL(783); break;
		case 784: LAUNCH_KERNEL(784); break;
		case 785: LAUNCH_KERNEL(785); break;
		case 786: LAUNCH_KERNEL(786); break;
		case 787: LAUNCH_KERNEL(787); break;
		case 788: LAUNCH_KERNEL(788); break;
		case 789: LAUNCH_KERNEL(789); break;
		case 790: LAUNCH_KERNEL(790); break;
		case 791: LAUNCH_KERNEL(791); break;
		case 792: LAUNCH_KERNEL(792); break;
		case 793: LAUNCH_KERNEL(793); break;
		case 794: LAUNCH_KERNEL(794); break;
		case 795: LAUNCH_KERNEL(795); break;
		case 796: LAUNCH_KERNEL(796); break;
		case 797: LAUNCH_KERNEL(797); break;
		case 798: LAUNCH_KERNEL(798); break;
		case 799: LAUNCH_KERNEL(799); break;
		case 800: LAUNCH_KERNEL(800); break;
		case 801: LAUNCH_KERNEL(801); break;
		case 802: LAUNCH_KERNEL(802); break;
		case 803: LAUNCH_KERNEL(803); break;
		case 804: LAUNCH_KERNEL(804); break;
		case 805: LAUNCH_KERNEL(805); break;
		case 806: LAUNCH_KERNEL(806); break;
		case 807: LAUNCH_KERNEL(807); break;
		case 808: LAUNCH_KERNEL(808); break;
		case 809: LAUNCH_KERNEL(809); break;
		case 810: LAUNCH_KERNEL(810); break;
		case 811: LAUNCH_KERNEL(811); break;
		case 812: LAUNCH_KERNEL(812); break;
		case 813: LAUNCH_KERNEL(813); break;
		case 814: LAUNCH_KERNEL(814); break;
		case 815: LAUNCH_KERNEL(815); break;
		case 816: LAUNCH_KERNEL(816); break;
		case 817: LAUNCH_KERNEL(817); break;
		case 818: LAUNCH_KERNEL(818); break;
		case 819: LAUNCH_KERNEL(819); break;
		case 820: LAUNCH_KERNEL(820); break;
		case 821: LAUNCH_KERNEL(821); break;
		case 822: LAUNCH_KERNEL(822); break;
		case 823: LAUNCH_KERNEL(823); break;
		case 824: LAUNCH_KERNEL(824); break;
		case 825: LAUNCH_KERNEL(825); break;
		case 826: LAUNCH_KERNEL(826); break;
		case 827: LAUNCH_KERNEL(827); break;
		case 828: LAUNCH_KERNEL(828); break;
		case 829: LAUNCH_KERNEL(829); break;
		case 830: LAUNCH_KERNEL(830); break;
		case 831: LAUNCH_KERNEL(831); break;
		case 832: LAUNCH_KERNEL(832); break;
		case 833: LAUNCH_KERNEL(833); break;
		case 834: LAUNCH_KERNEL(834); break;
		case 835: LAUNCH_KERNEL(835); break;
		case 836: LAUNCH_KERNEL(836); break;
		case 837: LAUNCH_KERNEL(837); break;
		case 838: LAUNCH_KERNEL(838); break;
		case 839: LAUNCH_KERNEL(839); break;
		case 840: LAUNCH_KERNEL(840); break;
		case 841: LAUNCH_KERNEL(841); break;
		case 842: LAUNCH_KERNEL(842); break;
		case 843: LAUNCH_KERNEL(843); break;
		case 844: LAUNCH_KERNEL(844); break;
		case 845: LAUNCH_KERNEL(845); break;
		case 846: LAUNCH_KERNEL(846); break;
		case 847: LAUNCH_KERNEL(847); break;
		case 848: LAUNCH_KERNEL(848); break;
		case 849: LAUNCH_KERNEL(849); break;
		case 850: LAUNCH_KERNEL(850); break;
		case 851: LAUNCH_KERNEL(851); break;
		case 852: LAUNCH_KERNEL(852); break;
		case 853: LAUNCH_KERNEL(853); break;
		case 854: LAUNCH_KERNEL(854); break;
		case 855: LAUNCH_KERNEL(855); break;
		case 856: LAUNCH_KERNEL(856); break;
		case 857: LAUNCH_KERNEL(857); break;
		case 858: LAUNCH_KERNEL(858); break;
		case 859: LAUNCH_KERNEL(859); break;
		case 860: LAUNCH_KERNEL(860); break;
		case 861: LAUNCH_KERNEL(861); break;
		case 862: LAUNCH_KERNEL(862); break;
		case 863: LAUNCH_KERNEL(863); break;
		case 864: LAUNCH_KERNEL(864); break;
		case 865: LAUNCH_KERNEL(865); break;
		case 866: LAUNCH_KERNEL(866); break;
		case 867: LAUNCH_KERNEL(867); break;
		case 868: LAUNCH_KERNEL(868); break;
		case 869: LAUNCH_KERNEL(869); break;
		case 870: LAUNCH_KERNEL(870); break;
		case 871: LAUNCH_KERNEL(871); break;
		case 872: LAUNCH_KERNEL(872); break;
		case 873: LAUNCH_KERNEL(873); break;
		case 874: LAUNCH_KERNEL(874); break;
		case 875: LAUNCH_KERNEL(875); break;
		case 876: LAUNCH_KERNEL(876); break;
		case 877: LAUNCH_KERNEL(877); break;
		case 878: LAUNCH_KERNEL(878); break;
		case 879: LAUNCH_KERNEL(879); break;
		case 880: LAUNCH_KERNEL(880); break;
		case 881: LAUNCH_KERNEL(881); break;
		case 882: LAUNCH_KERNEL(882); break;
		case 883: LAUNCH_KERNEL(883); break;
		case 884: LAUNCH_KERNEL(884); break;
		case 885: LAUNCH_KERNEL(885); break;
		case 886: LAUNCH_KERNEL(886); break;
		case 887: LAUNCH_KERNEL(887); break;
		case 888: LAUNCH_KERNEL(888); break;
		case 889: LAUNCH_KERNEL(889); break;
		case 890: LAUNCH_KERNEL(890); break;
		case 891: LAUNCH_KERNEL(891); break;
		case 892: LAUNCH_KERNEL(892); break;
		case 893: LAUNCH_KERNEL(893); break;
		case 894: LAUNCH_KERNEL(894); break;
		case 895: LAUNCH_KERNEL(895); break;
		case 896: LAUNCH_KERNEL(896); break;
		case 897: LAUNCH_KERNEL(897); break;
		case 898: LAUNCH_KERNEL(898); break;
		case 899: LAUNCH_KERNEL(899); break;
		case 900: LAUNCH_KERNEL(900); break;
		case 901: LAUNCH_KERNEL(901); break;
		case 902: LAUNCH_KERNEL(902); break;
		case 903: LAUNCH_KERNEL(903); break;
		case 904: LAUNCH_KERNEL(904); break;
		case 905: LAUNCH_KERNEL(905); break;
		case 906: LAUNCH_KERNEL(906); break;
		case 907: LAUNCH_KERNEL(907); break;
		case 908: LAUNCH_KERNEL(908); break;
		case 909: LAUNCH_KERNEL(909); break;
		case 910: LAUNCH_KERNEL(910); break;
		case 911: LAUNCH_KERNEL(911); break;
		case 912: LAUNCH_KERNEL(912); break;
		case 913: LAUNCH_KERNEL(913); break;
		case 914: LAUNCH_KERNEL(914); break;
		case 915: LAUNCH_KERNEL(915); break;
		case 916: LAUNCH_KERNEL(916); break;
		case 917: LAUNCH_KERNEL(917); break;
		case 918: LAUNCH_KERNEL(918); break;
		case 919: LAUNCH_KERNEL(919); break;
		case 920: LAUNCH_KERNEL(920); break;
		case 921: LAUNCH_KERNEL(921); break;
		case 922: LAUNCH_KERNEL(922); break;
		case 923: LAUNCH_KERNEL(923); break;
		case 924: LAUNCH_KERNEL(924); break;
		case 925: LAUNCH_KERNEL(925); break;
		case 926: LAUNCH_KERNEL(926); break;
		case 927: LAUNCH_KERNEL(927); break;
		case 928: LAUNCH_KERNEL(928); break;
		case 929: LAUNCH_KERNEL(929); break;
		case 930: LAUNCH_KERNEL(930); break;
		case 931: LAUNCH_KERNEL(931); break;
		case 932: LAUNCH_KERNEL(932); break;
		case 933: LAUNCH_KERNEL(933); break;
		case 934: LAUNCH_KERNEL(934); break;
		case 935: LAUNCH_KERNEL(935); break;
		case 936: LAUNCH_KERNEL(936); break;
		case 937: LAUNCH_KERNEL(937); break;
		case 938: LAUNCH_KERNEL(938); break;
		case 939: LAUNCH_KERNEL(939); break;
		case 940: LAUNCH_KERNEL(940); break;
		case 941: LAUNCH_KERNEL(941); break;
		case 942: LAUNCH_KERNEL(942); break;
		case 943: LAUNCH_KERNEL(943); break;
		case 944: LAUNCH_KERNEL(944); break;
		case 945: LAUNCH_KERNEL(945); break;
		case 946: LAUNCH_KERNEL(946); break;
		case 947: LAUNCH_KERNEL(947); break;
		case 948: LAUNCH_KERNEL(948); break;
		case 949: LAUNCH_KERNEL(949); break;
		case 950: LAUNCH_KERNEL(950); break;
		case 951: LAUNCH_KERNEL(951); break;
		case 952: LAUNCH_KERNEL(952); break;
		case 953: LAUNCH_KERNEL(953); break;
		case 954: LAUNCH_KERNEL(954); break;
		case 955: LAUNCH_KERNEL(955); break;
		case 956: LAUNCH_KERNEL(956); break;
		case 957: LAUNCH_KERNEL(957); break;
		case 958: LAUNCH_KERNEL(958); break;
		case 959: LAUNCH_KERNEL(959); break;
		case 960: LAUNCH_KERNEL(960); break;
		case 961: LAUNCH_KERNEL(961); break;
		case 962: LAUNCH_KERNEL(962); break;
		case 963: LAUNCH_KERNEL(963); break;
		case 964: LAUNCH_KERNEL(964); break;
		case 965: LAUNCH_KERNEL(965); break;
		case 966: LAUNCH_KERNEL(966); break;
		case 967: LAUNCH_KERNEL(967); break;
		case 968: LAUNCH_KERNEL(968); break;
		case 969: LAUNCH_KERNEL(969); break;
		case 970: LAUNCH_KERNEL(970); break;
		case 971: LAUNCH_KERNEL(971); break;
		case 972: LAUNCH_KERNEL(972); break;
		case 973: LAUNCH_KERNEL(973); break;
		case 974: LAUNCH_KERNEL(974); break;
		case 975: LAUNCH_KERNEL(975); break;
		case 976: LAUNCH_KERNEL(976); break;
		case 977: LAUNCH_KERNEL(977); break;
		case 978: LAUNCH_KERNEL(978); break;
		case 979: LAUNCH_KERNEL(979); break;
		case 980: LAUNCH_KERNEL(980); break;
		case 981: LAUNCH_KERNEL(981); break;
		case 982: LAUNCH_KERNEL(982); break;
		case 983: LAUNCH_KERNEL(983); break;
		case 984: LAUNCH_KERNEL(984); break;
		case 985: LAUNCH_KERNEL(985); break;
		case 986: LAUNCH_KERNEL(986); break;
		case 987: LAUNCH_KERNEL(987); break;
		case 988: LAUNCH_KERNEL(988); break;
		case 989: LAUNCH_KERNEL(989); break;
		case 990: LAUNCH_KERNEL(990); break;
		case 991: LAUNCH_KERNEL(991); break;
		case 992: LAUNCH_KERNEL(992); break;
		case 993: LAUNCH_KERNEL(993); break;
		case 994: LAUNCH_KERNEL(994); break;
		case 995: LAUNCH_KERNEL(995); break;
		case 996: LAUNCH_KERNEL(996); break;
		case 997: LAUNCH_KERNEL(997); break;
		case 998: LAUNCH_KERNEL(998); break;
		case 999: LAUNCH_KERNEL(999); break;
		case 1000: LAUNCH_KERNEL(1000); break;
		case 1001: LAUNCH_KERNEL(1001); break;
		case 1002: LAUNCH_KERNEL(1002); break;
		case 1003: LAUNCH_KERNEL(1003); break;
		case 1004: LAUNCH_KERNEL(1004); break;
		case 1005: LAUNCH_KERNEL(1005); break;
		case 1006: LAUNCH_KERNEL(1006); break;
		case 1007: LAUNCH_KERNEL(1007); break;
		case 1008: LAUNCH_KERNEL(1008); break;
		case 1009: LAUNCH_KERNEL(1009); break;
		case 1010: LAUNCH_KERNEL(1010); break;
		case 1011: LAUNCH_KERNEL(1011); break;
		case 1012: LAUNCH_KERNEL(1012); break;
		case 1013: LAUNCH_KERNEL(1013); break;
		case 1014: LAUNCH_KERNEL(1014); break;
		case 1015: LAUNCH_KERNEL(1015); break;
		case 1016: LAUNCH_KERNEL(1016); break;
		case 1017: LAUNCH_KERNEL(1017); break;
		case 1018: LAUNCH_KERNEL(1018); break;
		case 1019: LAUNCH_KERNEL(1019); break;
		case 1020: LAUNCH_KERNEL(1020); break;
		case 1021: LAUNCH_KERNEL(1021); break;
		case 1022: LAUNCH_KERNEL(1022); break;
		case 1023: LAUNCH_KERNEL(1023); break;
		default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
