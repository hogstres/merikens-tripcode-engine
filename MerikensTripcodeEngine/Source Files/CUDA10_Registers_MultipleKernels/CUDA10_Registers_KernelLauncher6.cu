// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 1536
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1537
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1538
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1539
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1540
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1541
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1542
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1543
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1544
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1545
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1546
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1547
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1548
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1549
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1550
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1551
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1552
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1553
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1554
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1555
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1556
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1557
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1558
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1559
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1560
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1561
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1562
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1563
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1564
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1565
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1566
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1567
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1568
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1569
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1570
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1571
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1572
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1573
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1574
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1575
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1576
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1577
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1578
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1579
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1580
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1581
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1582
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1583
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1584
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1585
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1586
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1587
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1588
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1589
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1590
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1591
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1592
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1593
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1594
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1595
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1596
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1597
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1598
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1599
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1600
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1601
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1602
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1603
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1604
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1605
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1606
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1607
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1608
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1609
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1610
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1611
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1612
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1613
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1614
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1615
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1616
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1617
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1618
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1619
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1620
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1621
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1622
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1623
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1624
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1625
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1626
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1627
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1628
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1629
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1630
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1631
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1632
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1633
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1634
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1635
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1636
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1637
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1638
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1639
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1640
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1641
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1642
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1643
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1644
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1645
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1646
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1647
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1648
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1649
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1650
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1651
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1652
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1653
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1654
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1655
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1656
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1657
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1658
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1659
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1660
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1661
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1662
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1663
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1664
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1665
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1666
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1667
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1668
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1669
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1670
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1671
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1672
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1673
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1674
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1675
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1676
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1677
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1678
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1679
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1680
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1681
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1682
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1683
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1684
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1685
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1686
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1687
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1688
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1689
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1690
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1691
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1692
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1693
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1694
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1695
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1696
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1697
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1698
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1699
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1700
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1701
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1702
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1703
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1704
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1705
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1706
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1707
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1708
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1709
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1710
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1711
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1712
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1713
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1714
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1715
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1716
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1717
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1718
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1719
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1720
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1721
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1722
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1723
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1724
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1725
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1726
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1727
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1728
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1729
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1730
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1731
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1732
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1733
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1734
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1735
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1736
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1737
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1738
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1739
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1740
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1741
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1742
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1743
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1744
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1745
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1746
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1747
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1748
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1749
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1750
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1751
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1752
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1753
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1754
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1755
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1756
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1757
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1758
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1759
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1760
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1761
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1762
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1763
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1764
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1765
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1766
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1767
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1768
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1769
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1770
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1771
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1772
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1773
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1774
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1775
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1776
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1777
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1778
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1779
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1780
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1781
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1782
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1783
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1784
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1785
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1786
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1787
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1788
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1789
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1790
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1791
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher6()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel6(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 1536: LAUNCH_KERNEL(1536); break;
	case 1537: LAUNCH_KERNEL(1537); break;
	case 1538: LAUNCH_KERNEL(1538); break;
	case 1539: LAUNCH_KERNEL(1539); break;
	case 1540: LAUNCH_KERNEL(1540); break;
	case 1541: LAUNCH_KERNEL(1541); break;
	case 1542: LAUNCH_KERNEL(1542); break;
	case 1543: LAUNCH_KERNEL(1543); break;
	case 1544: LAUNCH_KERNEL(1544); break;
	case 1545: LAUNCH_KERNEL(1545); break;
	case 1546: LAUNCH_KERNEL(1546); break;
	case 1547: LAUNCH_KERNEL(1547); break;
	case 1548: LAUNCH_KERNEL(1548); break;
	case 1549: LAUNCH_KERNEL(1549); break;
	case 1550: LAUNCH_KERNEL(1550); break;
	case 1551: LAUNCH_KERNEL(1551); break;
	case 1552: LAUNCH_KERNEL(1552); break;
	case 1553: LAUNCH_KERNEL(1553); break;
	case 1554: LAUNCH_KERNEL(1554); break;
	case 1555: LAUNCH_KERNEL(1555); break;
	case 1556: LAUNCH_KERNEL(1556); break;
	case 1557: LAUNCH_KERNEL(1557); break;
	case 1558: LAUNCH_KERNEL(1558); break;
	case 1559: LAUNCH_KERNEL(1559); break;
	case 1560: LAUNCH_KERNEL(1560); break;
	case 1561: LAUNCH_KERNEL(1561); break;
	case 1562: LAUNCH_KERNEL(1562); break;
	case 1563: LAUNCH_KERNEL(1563); break;
	case 1564: LAUNCH_KERNEL(1564); break;
	case 1565: LAUNCH_KERNEL(1565); break;
	case 1566: LAUNCH_KERNEL(1566); break;
	case 1567: LAUNCH_KERNEL(1567); break;
	case 1568: LAUNCH_KERNEL(1568); break;
	case 1569: LAUNCH_KERNEL(1569); break;
	case 1570: LAUNCH_KERNEL(1570); break;
	case 1571: LAUNCH_KERNEL(1571); break;
	case 1572: LAUNCH_KERNEL(1572); break;
	case 1573: LAUNCH_KERNEL(1573); break;
	case 1574: LAUNCH_KERNEL(1574); break;
	case 1575: LAUNCH_KERNEL(1575); break;
	case 1576: LAUNCH_KERNEL(1576); break;
	case 1577: LAUNCH_KERNEL(1577); break;
	case 1578: LAUNCH_KERNEL(1578); break;
	case 1579: LAUNCH_KERNEL(1579); break;
	case 1580: LAUNCH_KERNEL(1580); break;
	case 1581: LAUNCH_KERNEL(1581); break;
	case 1582: LAUNCH_KERNEL(1582); break;
	case 1583: LAUNCH_KERNEL(1583); break;
	case 1584: LAUNCH_KERNEL(1584); break;
	case 1585: LAUNCH_KERNEL(1585); break;
	case 1586: LAUNCH_KERNEL(1586); break;
	case 1587: LAUNCH_KERNEL(1587); break;
	case 1588: LAUNCH_KERNEL(1588); break;
	case 1589: LAUNCH_KERNEL(1589); break;
	case 1590: LAUNCH_KERNEL(1590); break;
	case 1591: LAUNCH_KERNEL(1591); break;
	case 1592: LAUNCH_KERNEL(1592); break;
	case 1593: LAUNCH_KERNEL(1593); break;
	case 1594: LAUNCH_KERNEL(1594); break;
	case 1595: LAUNCH_KERNEL(1595); break;
	case 1596: LAUNCH_KERNEL(1596); break;
	case 1597: LAUNCH_KERNEL(1597); break;
	case 1598: LAUNCH_KERNEL(1598); break;
	case 1599: LAUNCH_KERNEL(1599); break;
	case 1600: LAUNCH_KERNEL(1600); break;
	case 1601: LAUNCH_KERNEL(1601); break;
	case 1602: LAUNCH_KERNEL(1602); break;
	case 1603: LAUNCH_KERNEL(1603); break;
	case 1604: LAUNCH_KERNEL(1604); break;
	case 1605: LAUNCH_KERNEL(1605); break;
	case 1606: LAUNCH_KERNEL(1606); break;
	case 1607: LAUNCH_KERNEL(1607); break;
	case 1608: LAUNCH_KERNEL(1608); break;
	case 1609: LAUNCH_KERNEL(1609); break;
	case 1610: LAUNCH_KERNEL(1610); break;
	case 1611: LAUNCH_KERNEL(1611); break;
	case 1612: LAUNCH_KERNEL(1612); break;
	case 1613: LAUNCH_KERNEL(1613); break;
	case 1614: LAUNCH_KERNEL(1614); break;
	case 1615: LAUNCH_KERNEL(1615); break;
	case 1616: LAUNCH_KERNEL(1616); break;
	case 1617: LAUNCH_KERNEL(1617); break;
	case 1618: LAUNCH_KERNEL(1618); break;
	case 1619: LAUNCH_KERNEL(1619); break;
	case 1620: LAUNCH_KERNEL(1620); break;
	case 1621: LAUNCH_KERNEL(1621); break;
	case 1622: LAUNCH_KERNEL(1622); break;
	case 1623: LAUNCH_KERNEL(1623); break;
	case 1624: LAUNCH_KERNEL(1624); break;
	case 1625: LAUNCH_KERNEL(1625); break;
	case 1626: LAUNCH_KERNEL(1626); break;
	case 1627: LAUNCH_KERNEL(1627); break;
	case 1628: LAUNCH_KERNEL(1628); break;
	case 1629: LAUNCH_KERNEL(1629); break;
	case 1630: LAUNCH_KERNEL(1630); break;
	case 1631: LAUNCH_KERNEL(1631); break;
	case 1632: LAUNCH_KERNEL(1632); break;
	case 1633: LAUNCH_KERNEL(1633); break;
	case 1634: LAUNCH_KERNEL(1634); break;
	case 1635: LAUNCH_KERNEL(1635); break;
	case 1636: LAUNCH_KERNEL(1636); break;
	case 1637: LAUNCH_KERNEL(1637); break;
	case 1638: LAUNCH_KERNEL(1638); break;
	case 1639: LAUNCH_KERNEL(1639); break;
	case 1640: LAUNCH_KERNEL(1640); break;
	case 1641: LAUNCH_KERNEL(1641); break;
	case 1642: LAUNCH_KERNEL(1642); break;
	case 1643: LAUNCH_KERNEL(1643); break;
	case 1644: LAUNCH_KERNEL(1644); break;
	case 1645: LAUNCH_KERNEL(1645); break;
	case 1646: LAUNCH_KERNEL(1646); break;
	case 1647: LAUNCH_KERNEL(1647); break;
	case 1648: LAUNCH_KERNEL(1648); break;
	case 1649: LAUNCH_KERNEL(1649); break;
	case 1650: LAUNCH_KERNEL(1650); break;
	case 1651: LAUNCH_KERNEL(1651); break;
	case 1652: LAUNCH_KERNEL(1652); break;
	case 1653: LAUNCH_KERNEL(1653); break;
	case 1654: LAUNCH_KERNEL(1654); break;
	case 1655: LAUNCH_KERNEL(1655); break;
	case 1656: LAUNCH_KERNEL(1656); break;
	case 1657: LAUNCH_KERNEL(1657); break;
	case 1658: LAUNCH_KERNEL(1658); break;
	case 1659: LAUNCH_KERNEL(1659); break;
	case 1660: LAUNCH_KERNEL(1660); break;
	case 1661: LAUNCH_KERNEL(1661); break;
	case 1662: LAUNCH_KERNEL(1662); break;
	case 1663: LAUNCH_KERNEL(1663); break;
	case 1664: LAUNCH_KERNEL(1664); break;
	case 1665: LAUNCH_KERNEL(1665); break;
	case 1666: LAUNCH_KERNEL(1666); break;
	case 1667: LAUNCH_KERNEL(1667); break;
	case 1668: LAUNCH_KERNEL(1668); break;
	case 1669: LAUNCH_KERNEL(1669); break;
	case 1670: LAUNCH_KERNEL(1670); break;
	case 1671: LAUNCH_KERNEL(1671); break;
	case 1672: LAUNCH_KERNEL(1672); break;
	case 1673: LAUNCH_KERNEL(1673); break;
	case 1674: LAUNCH_KERNEL(1674); break;
	case 1675: LAUNCH_KERNEL(1675); break;
	case 1676: LAUNCH_KERNEL(1676); break;
	case 1677: LAUNCH_KERNEL(1677); break;
	case 1678: LAUNCH_KERNEL(1678); break;
	case 1679: LAUNCH_KERNEL(1679); break;
	case 1680: LAUNCH_KERNEL(1680); break;
	case 1681: LAUNCH_KERNEL(1681); break;
	case 1682: LAUNCH_KERNEL(1682); break;
	case 1683: LAUNCH_KERNEL(1683); break;
	case 1684: LAUNCH_KERNEL(1684); break;
	case 1685: LAUNCH_KERNEL(1685); break;
	case 1686: LAUNCH_KERNEL(1686); break;
	case 1687: LAUNCH_KERNEL(1687); break;
	case 1688: LAUNCH_KERNEL(1688); break;
	case 1689: LAUNCH_KERNEL(1689); break;
	case 1690: LAUNCH_KERNEL(1690); break;
	case 1691: LAUNCH_KERNEL(1691); break;
	case 1692: LAUNCH_KERNEL(1692); break;
	case 1693: LAUNCH_KERNEL(1693); break;
	case 1694: LAUNCH_KERNEL(1694); break;
	case 1695: LAUNCH_KERNEL(1695); break;
	case 1696: LAUNCH_KERNEL(1696); break;
	case 1697: LAUNCH_KERNEL(1697); break;
	case 1698: LAUNCH_KERNEL(1698); break;
	case 1699: LAUNCH_KERNEL(1699); break;
	case 1700: LAUNCH_KERNEL(1700); break;
	case 1701: LAUNCH_KERNEL(1701); break;
	case 1702: LAUNCH_KERNEL(1702); break;
	case 1703: LAUNCH_KERNEL(1703); break;
	case 1704: LAUNCH_KERNEL(1704); break;
	case 1705: LAUNCH_KERNEL(1705); break;
	case 1706: LAUNCH_KERNEL(1706); break;
	case 1707: LAUNCH_KERNEL(1707); break;
	case 1708: LAUNCH_KERNEL(1708); break;
	case 1709: LAUNCH_KERNEL(1709); break;
	case 1710: LAUNCH_KERNEL(1710); break;
	case 1711: LAUNCH_KERNEL(1711); break;
	case 1712: LAUNCH_KERNEL(1712); break;
	case 1713: LAUNCH_KERNEL(1713); break;
	case 1714: LAUNCH_KERNEL(1714); break;
	case 1715: LAUNCH_KERNEL(1715); break;
	case 1716: LAUNCH_KERNEL(1716); break;
	case 1717: LAUNCH_KERNEL(1717); break;
	case 1718: LAUNCH_KERNEL(1718); break;
	case 1719: LAUNCH_KERNEL(1719); break;
	case 1720: LAUNCH_KERNEL(1720); break;
	case 1721: LAUNCH_KERNEL(1721); break;
	case 1722: LAUNCH_KERNEL(1722); break;
	case 1723: LAUNCH_KERNEL(1723); break;
	case 1724: LAUNCH_KERNEL(1724); break;
	case 1725: LAUNCH_KERNEL(1725); break;
	case 1726: LAUNCH_KERNEL(1726); break;
	case 1727: LAUNCH_KERNEL(1727); break;
	case 1728: LAUNCH_KERNEL(1728); break;
	case 1729: LAUNCH_KERNEL(1729); break;
	case 1730: LAUNCH_KERNEL(1730); break;
	case 1731: LAUNCH_KERNEL(1731); break;
	case 1732: LAUNCH_KERNEL(1732); break;
	case 1733: LAUNCH_KERNEL(1733); break;
	case 1734: LAUNCH_KERNEL(1734); break;
	case 1735: LAUNCH_KERNEL(1735); break;
	case 1736: LAUNCH_KERNEL(1736); break;
	case 1737: LAUNCH_KERNEL(1737); break;
	case 1738: LAUNCH_KERNEL(1738); break;
	case 1739: LAUNCH_KERNEL(1739); break;
	case 1740: LAUNCH_KERNEL(1740); break;
	case 1741: LAUNCH_KERNEL(1741); break;
	case 1742: LAUNCH_KERNEL(1742); break;
	case 1743: LAUNCH_KERNEL(1743); break;
	case 1744: LAUNCH_KERNEL(1744); break;
	case 1745: LAUNCH_KERNEL(1745); break;
	case 1746: LAUNCH_KERNEL(1746); break;
	case 1747: LAUNCH_KERNEL(1747); break;
	case 1748: LAUNCH_KERNEL(1748); break;
	case 1749: LAUNCH_KERNEL(1749); break;
	case 1750: LAUNCH_KERNEL(1750); break;
	case 1751: LAUNCH_KERNEL(1751); break;
	case 1752: LAUNCH_KERNEL(1752); break;
	case 1753: LAUNCH_KERNEL(1753); break;
	case 1754: LAUNCH_KERNEL(1754); break;
	case 1755: LAUNCH_KERNEL(1755); break;
	case 1756: LAUNCH_KERNEL(1756); break;
	case 1757: LAUNCH_KERNEL(1757); break;
	case 1758: LAUNCH_KERNEL(1758); break;
	case 1759: LAUNCH_KERNEL(1759); break;
	case 1760: LAUNCH_KERNEL(1760); break;
	case 1761: LAUNCH_KERNEL(1761); break;
	case 1762: LAUNCH_KERNEL(1762); break;
	case 1763: LAUNCH_KERNEL(1763); break;
	case 1764: LAUNCH_KERNEL(1764); break;
	case 1765: LAUNCH_KERNEL(1765); break;
	case 1766: LAUNCH_KERNEL(1766); break;
	case 1767: LAUNCH_KERNEL(1767); break;
	case 1768: LAUNCH_KERNEL(1768); break;
	case 1769: LAUNCH_KERNEL(1769); break;
	case 1770: LAUNCH_KERNEL(1770); break;
	case 1771: LAUNCH_KERNEL(1771); break;
	case 1772: LAUNCH_KERNEL(1772); break;
	case 1773: LAUNCH_KERNEL(1773); break;
	case 1774: LAUNCH_KERNEL(1774); break;
	case 1775: LAUNCH_KERNEL(1775); break;
	case 1776: LAUNCH_KERNEL(1776); break;
	case 1777: LAUNCH_KERNEL(1777); break;
	case 1778: LAUNCH_KERNEL(1778); break;
	case 1779: LAUNCH_KERNEL(1779); break;
	case 1780: LAUNCH_KERNEL(1780); break;
	case 1781: LAUNCH_KERNEL(1781); break;
	case 1782: LAUNCH_KERNEL(1782); break;
	case 1783: LAUNCH_KERNEL(1783); break;
	case 1784: LAUNCH_KERNEL(1784); break;
	case 1785: LAUNCH_KERNEL(1785); break;
	case 1786: LAUNCH_KERNEL(1786); break;
	case 1787: LAUNCH_KERNEL(1787); break;
	case 1788: LAUNCH_KERNEL(1788); break;
	case 1789: LAUNCH_KERNEL(1789); break;
	case 1790: LAUNCH_KERNEL(1790); break;
	case 1791: LAUNCH_KERNEL(1791); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
