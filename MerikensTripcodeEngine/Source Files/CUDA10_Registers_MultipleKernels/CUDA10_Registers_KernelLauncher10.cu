// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 2560
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2561
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2562
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2563
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2564
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2565
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2566
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2567
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2568
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2569
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2570
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2571
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2572
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2573
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2574
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2575
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2576
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2577
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2578
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2579
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2580
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2581
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2582
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2583
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2584
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2585
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2586
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2587
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2588
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2589
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2590
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2591
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2592
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2593
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2594
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2595
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2596
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2597
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2598
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2599
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2600
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2601
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2602
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2603
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2604
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2605
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2606
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2607
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2608
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2609
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2610
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2611
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2612
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2613
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2614
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2615
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2616
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2617
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2618
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2619
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2620
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2621
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2622
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2623
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2624
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2625
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2626
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2627
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2628
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2629
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2630
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2631
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2632
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2633
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2634
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2635
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2636
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2637
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2638
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2639
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2640
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2641
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2642
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2643
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2644
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2645
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2646
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2647
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2648
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2649
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2650
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2651
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2652
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2653
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2654
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2655
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2656
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2657
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2658
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2659
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2660
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2661
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2662
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2663
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2664
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2665
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2666
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2667
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2668
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2669
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2670
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2671
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2672
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2673
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2674
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2675
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2676
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2677
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2678
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2679
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2680
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2681
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2682
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2683
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2684
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2685
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2686
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2687
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2688
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2689
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2690
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2691
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2692
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2693
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2694
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2695
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2696
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2697
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2698
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2699
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2700
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2701
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2702
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2703
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2704
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2705
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2706
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2707
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2708
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2709
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2710
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2711
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2712
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2713
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2714
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2715
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2716
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2717
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2718
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2719
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2720
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2721
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2722
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2723
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2724
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2725
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2726
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2727
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2728
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2729
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2730
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2731
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2732
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2733
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2734
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2735
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2736
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2737
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2738
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2739
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2740
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2741
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2742
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2743
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2744
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2745
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2746
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2747
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2748
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2749
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2750
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2751
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2752
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2753
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2754
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2755
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2756
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2757
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2758
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2759
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2760
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2761
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2762
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2763
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2764
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2765
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2766
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2767
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2768
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2769
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2770
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2771
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2772
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2773
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2774
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2775
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2776
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2777
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2778
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2779
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2780
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2781
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2782
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2783
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2784
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2785
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2786
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2787
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2788
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2789
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2790
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2791
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2792
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2793
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2794
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2795
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2796
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2797
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2798
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2799
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2800
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2801
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2802
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2803
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2804
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2805
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2806
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2807
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2808
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2809
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2810
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2811
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2812
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2813
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2814
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2815
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher10()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel10(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 2560: LAUNCH_KERNEL(2560); break;
	case 2561: LAUNCH_KERNEL(2561); break;
	case 2562: LAUNCH_KERNEL(2562); break;
	case 2563: LAUNCH_KERNEL(2563); break;
	case 2564: LAUNCH_KERNEL(2564); break;
	case 2565: LAUNCH_KERNEL(2565); break;
	case 2566: LAUNCH_KERNEL(2566); break;
	case 2567: LAUNCH_KERNEL(2567); break;
	case 2568: LAUNCH_KERNEL(2568); break;
	case 2569: LAUNCH_KERNEL(2569); break;
	case 2570: LAUNCH_KERNEL(2570); break;
	case 2571: LAUNCH_KERNEL(2571); break;
	case 2572: LAUNCH_KERNEL(2572); break;
	case 2573: LAUNCH_KERNEL(2573); break;
	case 2574: LAUNCH_KERNEL(2574); break;
	case 2575: LAUNCH_KERNEL(2575); break;
	case 2576: LAUNCH_KERNEL(2576); break;
	case 2577: LAUNCH_KERNEL(2577); break;
	case 2578: LAUNCH_KERNEL(2578); break;
	case 2579: LAUNCH_KERNEL(2579); break;
	case 2580: LAUNCH_KERNEL(2580); break;
	case 2581: LAUNCH_KERNEL(2581); break;
	case 2582: LAUNCH_KERNEL(2582); break;
	case 2583: LAUNCH_KERNEL(2583); break;
	case 2584: LAUNCH_KERNEL(2584); break;
	case 2585: LAUNCH_KERNEL(2585); break;
	case 2586: LAUNCH_KERNEL(2586); break;
	case 2587: LAUNCH_KERNEL(2587); break;
	case 2588: LAUNCH_KERNEL(2588); break;
	case 2589: LAUNCH_KERNEL(2589); break;
	case 2590: LAUNCH_KERNEL(2590); break;
	case 2591: LAUNCH_KERNEL(2591); break;
	case 2592: LAUNCH_KERNEL(2592); break;
	case 2593: LAUNCH_KERNEL(2593); break;
	case 2594: LAUNCH_KERNEL(2594); break;
	case 2595: LAUNCH_KERNEL(2595); break;
	case 2596: LAUNCH_KERNEL(2596); break;
	case 2597: LAUNCH_KERNEL(2597); break;
	case 2598: LAUNCH_KERNEL(2598); break;
	case 2599: LAUNCH_KERNEL(2599); break;
	case 2600: LAUNCH_KERNEL(2600); break;
	case 2601: LAUNCH_KERNEL(2601); break;
	case 2602: LAUNCH_KERNEL(2602); break;
	case 2603: LAUNCH_KERNEL(2603); break;
	case 2604: LAUNCH_KERNEL(2604); break;
	case 2605: LAUNCH_KERNEL(2605); break;
	case 2606: LAUNCH_KERNEL(2606); break;
	case 2607: LAUNCH_KERNEL(2607); break;
	case 2608: LAUNCH_KERNEL(2608); break;
	case 2609: LAUNCH_KERNEL(2609); break;
	case 2610: LAUNCH_KERNEL(2610); break;
	case 2611: LAUNCH_KERNEL(2611); break;
	case 2612: LAUNCH_KERNEL(2612); break;
	case 2613: LAUNCH_KERNEL(2613); break;
	case 2614: LAUNCH_KERNEL(2614); break;
	case 2615: LAUNCH_KERNEL(2615); break;
	case 2616: LAUNCH_KERNEL(2616); break;
	case 2617: LAUNCH_KERNEL(2617); break;
	case 2618: LAUNCH_KERNEL(2618); break;
	case 2619: LAUNCH_KERNEL(2619); break;
	case 2620: LAUNCH_KERNEL(2620); break;
	case 2621: LAUNCH_KERNEL(2621); break;
	case 2622: LAUNCH_KERNEL(2622); break;
	case 2623: LAUNCH_KERNEL(2623); break;
	case 2624: LAUNCH_KERNEL(2624); break;
	case 2625: LAUNCH_KERNEL(2625); break;
	case 2626: LAUNCH_KERNEL(2626); break;
	case 2627: LAUNCH_KERNEL(2627); break;
	case 2628: LAUNCH_KERNEL(2628); break;
	case 2629: LAUNCH_KERNEL(2629); break;
	case 2630: LAUNCH_KERNEL(2630); break;
	case 2631: LAUNCH_KERNEL(2631); break;
	case 2632: LAUNCH_KERNEL(2632); break;
	case 2633: LAUNCH_KERNEL(2633); break;
	case 2634: LAUNCH_KERNEL(2634); break;
	case 2635: LAUNCH_KERNEL(2635); break;
	case 2636: LAUNCH_KERNEL(2636); break;
	case 2637: LAUNCH_KERNEL(2637); break;
	case 2638: LAUNCH_KERNEL(2638); break;
	case 2639: LAUNCH_KERNEL(2639); break;
	case 2640: LAUNCH_KERNEL(2640); break;
	case 2641: LAUNCH_KERNEL(2641); break;
	case 2642: LAUNCH_KERNEL(2642); break;
	case 2643: LAUNCH_KERNEL(2643); break;
	case 2644: LAUNCH_KERNEL(2644); break;
	case 2645: LAUNCH_KERNEL(2645); break;
	case 2646: LAUNCH_KERNEL(2646); break;
	case 2647: LAUNCH_KERNEL(2647); break;
	case 2648: LAUNCH_KERNEL(2648); break;
	case 2649: LAUNCH_KERNEL(2649); break;
	case 2650: LAUNCH_KERNEL(2650); break;
	case 2651: LAUNCH_KERNEL(2651); break;
	case 2652: LAUNCH_KERNEL(2652); break;
	case 2653: LAUNCH_KERNEL(2653); break;
	case 2654: LAUNCH_KERNEL(2654); break;
	case 2655: LAUNCH_KERNEL(2655); break;
	case 2656: LAUNCH_KERNEL(2656); break;
	case 2657: LAUNCH_KERNEL(2657); break;
	case 2658: LAUNCH_KERNEL(2658); break;
	case 2659: LAUNCH_KERNEL(2659); break;
	case 2660: LAUNCH_KERNEL(2660); break;
	case 2661: LAUNCH_KERNEL(2661); break;
	case 2662: LAUNCH_KERNEL(2662); break;
	case 2663: LAUNCH_KERNEL(2663); break;
	case 2664: LAUNCH_KERNEL(2664); break;
	case 2665: LAUNCH_KERNEL(2665); break;
	case 2666: LAUNCH_KERNEL(2666); break;
	case 2667: LAUNCH_KERNEL(2667); break;
	case 2668: LAUNCH_KERNEL(2668); break;
	case 2669: LAUNCH_KERNEL(2669); break;
	case 2670: LAUNCH_KERNEL(2670); break;
	case 2671: LAUNCH_KERNEL(2671); break;
	case 2672: LAUNCH_KERNEL(2672); break;
	case 2673: LAUNCH_KERNEL(2673); break;
	case 2674: LAUNCH_KERNEL(2674); break;
	case 2675: LAUNCH_KERNEL(2675); break;
	case 2676: LAUNCH_KERNEL(2676); break;
	case 2677: LAUNCH_KERNEL(2677); break;
	case 2678: LAUNCH_KERNEL(2678); break;
	case 2679: LAUNCH_KERNEL(2679); break;
	case 2680: LAUNCH_KERNEL(2680); break;
	case 2681: LAUNCH_KERNEL(2681); break;
	case 2682: LAUNCH_KERNEL(2682); break;
	case 2683: LAUNCH_KERNEL(2683); break;
	case 2684: LAUNCH_KERNEL(2684); break;
	case 2685: LAUNCH_KERNEL(2685); break;
	case 2686: LAUNCH_KERNEL(2686); break;
	case 2687: LAUNCH_KERNEL(2687); break;
	case 2688: LAUNCH_KERNEL(2688); break;
	case 2689: LAUNCH_KERNEL(2689); break;
	case 2690: LAUNCH_KERNEL(2690); break;
	case 2691: LAUNCH_KERNEL(2691); break;
	case 2692: LAUNCH_KERNEL(2692); break;
	case 2693: LAUNCH_KERNEL(2693); break;
	case 2694: LAUNCH_KERNEL(2694); break;
	case 2695: LAUNCH_KERNEL(2695); break;
	case 2696: LAUNCH_KERNEL(2696); break;
	case 2697: LAUNCH_KERNEL(2697); break;
	case 2698: LAUNCH_KERNEL(2698); break;
	case 2699: LAUNCH_KERNEL(2699); break;
	case 2700: LAUNCH_KERNEL(2700); break;
	case 2701: LAUNCH_KERNEL(2701); break;
	case 2702: LAUNCH_KERNEL(2702); break;
	case 2703: LAUNCH_KERNEL(2703); break;
	case 2704: LAUNCH_KERNEL(2704); break;
	case 2705: LAUNCH_KERNEL(2705); break;
	case 2706: LAUNCH_KERNEL(2706); break;
	case 2707: LAUNCH_KERNEL(2707); break;
	case 2708: LAUNCH_KERNEL(2708); break;
	case 2709: LAUNCH_KERNEL(2709); break;
	case 2710: LAUNCH_KERNEL(2710); break;
	case 2711: LAUNCH_KERNEL(2711); break;
	case 2712: LAUNCH_KERNEL(2712); break;
	case 2713: LAUNCH_KERNEL(2713); break;
	case 2714: LAUNCH_KERNEL(2714); break;
	case 2715: LAUNCH_KERNEL(2715); break;
	case 2716: LAUNCH_KERNEL(2716); break;
	case 2717: LAUNCH_KERNEL(2717); break;
	case 2718: LAUNCH_KERNEL(2718); break;
	case 2719: LAUNCH_KERNEL(2719); break;
	case 2720: LAUNCH_KERNEL(2720); break;
	case 2721: LAUNCH_KERNEL(2721); break;
	case 2722: LAUNCH_KERNEL(2722); break;
	case 2723: LAUNCH_KERNEL(2723); break;
	case 2724: LAUNCH_KERNEL(2724); break;
	case 2725: LAUNCH_KERNEL(2725); break;
	case 2726: LAUNCH_KERNEL(2726); break;
	case 2727: LAUNCH_KERNEL(2727); break;
	case 2728: LAUNCH_KERNEL(2728); break;
	case 2729: LAUNCH_KERNEL(2729); break;
	case 2730: LAUNCH_KERNEL(2730); break;
	case 2731: LAUNCH_KERNEL(2731); break;
	case 2732: LAUNCH_KERNEL(2732); break;
	case 2733: LAUNCH_KERNEL(2733); break;
	case 2734: LAUNCH_KERNEL(2734); break;
	case 2735: LAUNCH_KERNEL(2735); break;
	case 2736: LAUNCH_KERNEL(2736); break;
	case 2737: LAUNCH_KERNEL(2737); break;
	case 2738: LAUNCH_KERNEL(2738); break;
	case 2739: LAUNCH_KERNEL(2739); break;
	case 2740: LAUNCH_KERNEL(2740); break;
	case 2741: LAUNCH_KERNEL(2741); break;
	case 2742: LAUNCH_KERNEL(2742); break;
	case 2743: LAUNCH_KERNEL(2743); break;
	case 2744: LAUNCH_KERNEL(2744); break;
	case 2745: LAUNCH_KERNEL(2745); break;
	case 2746: LAUNCH_KERNEL(2746); break;
	case 2747: LAUNCH_KERNEL(2747); break;
	case 2748: LAUNCH_KERNEL(2748); break;
	case 2749: LAUNCH_KERNEL(2749); break;
	case 2750: LAUNCH_KERNEL(2750); break;
	case 2751: LAUNCH_KERNEL(2751); break;
	case 2752: LAUNCH_KERNEL(2752); break;
	case 2753: LAUNCH_KERNEL(2753); break;
	case 2754: LAUNCH_KERNEL(2754); break;
	case 2755: LAUNCH_KERNEL(2755); break;
	case 2756: LAUNCH_KERNEL(2756); break;
	case 2757: LAUNCH_KERNEL(2757); break;
	case 2758: LAUNCH_KERNEL(2758); break;
	case 2759: LAUNCH_KERNEL(2759); break;
	case 2760: LAUNCH_KERNEL(2760); break;
	case 2761: LAUNCH_KERNEL(2761); break;
	case 2762: LAUNCH_KERNEL(2762); break;
	case 2763: LAUNCH_KERNEL(2763); break;
	case 2764: LAUNCH_KERNEL(2764); break;
	case 2765: LAUNCH_KERNEL(2765); break;
	case 2766: LAUNCH_KERNEL(2766); break;
	case 2767: LAUNCH_KERNEL(2767); break;
	case 2768: LAUNCH_KERNEL(2768); break;
	case 2769: LAUNCH_KERNEL(2769); break;
	case 2770: LAUNCH_KERNEL(2770); break;
	case 2771: LAUNCH_KERNEL(2771); break;
	case 2772: LAUNCH_KERNEL(2772); break;
	case 2773: LAUNCH_KERNEL(2773); break;
	case 2774: LAUNCH_KERNEL(2774); break;
	case 2775: LAUNCH_KERNEL(2775); break;
	case 2776: LAUNCH_KERNEL(2776); break;
	case 2777: LAUNCH_KERNEL(2777); break;
	case 2778: LAUNCH_KERNEL(2778); break;
	case 2779: LAUNCH_KERNEL(2779); break;
	case 2780: LAUNCH_KERNEL(2780); break;
	case 2781: LAUNCH_KERNEL(2781); break;
	case 2782: LAUNCH_KERNEL(2782); break;
	case 2783: LAUNCH_KERNEL(2783); break;
	case 2784: LAUNCH_KERNEL(2784); break;
	case 2785: LAUNCH_KERNEL(2785); break;
	case 2786: LAUNCH_KERNEL(2786); break;
	case 2787: LAUNCH_KERNEL(2787); break;
	case 2788: LAUNCH_KERNEL(2788); break;
	case 2789: LAUNCH_KERNEL(2789); break;
	case 2790: LAUNCH_KERNEL(2790); break;
	case 2791: LAUNCH_KERNEL(2791); break;
	case 2792: LAUNCH_KERNEL(2792); break;
	case 2793: LAUNCH_KERNEL(2793); break;
	case 2794: LAUNCH_KERNEL(2794); break;
	case 2795: LAUNCH_KERNEL(2795); break;
	case 2796: LAUNCH_KERNEL(2796); break;
	case 2797: LAUNCH_KERNEL(2797); break;
	case 2798: LAUNCH_KERNEL(2798); break;
	case 2799: LAUNCH_KERNEL(2799); break;
	case 2800: LAUNCH_KERNEL(2800); break;
	case 2801: LAUNCH_KERNEL(2801); break;
	case 2802: LAUNCH_KERNEL(2802); break;
	case 2803: LAUNCH_KERNEL(2803); break;
	case 2804: LAUNCH_KERNEL(2804); break;
	case 2805: LAUNCH_KERNEL(2805); break;
	case 2806: LAUNCH_KERNEL(2806); break;
	case 2807: LAUNCH_KERNEL(2807); break;
	case 2808: LAUNCH_KERNEL(2808); break;
	case 2809: LAUNCH_KERNEL(2809); break;
	case 2810: LAUNCH_KERNEL(2810); break;
	case 2811: LAUNCH_KERNEL(2811); break;
	case 2812: LAUNCH_KERNEL(2812); break;
	case 2813: LAUNCH_KERNEL(2813); break;
	case 2814: LAUNCH_KERNEL(2814); break;
	case 2815: LAUNCH_KERNEL(2815); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
