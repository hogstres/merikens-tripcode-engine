// Meriken's Tripcode Engine 2.0.1
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 1024
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1025
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1026
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1027
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1028
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1029
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1030
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1031
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1032
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1033
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1034
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1035
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1036
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1037
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1038
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1039
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1040
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1041
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1042
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1043
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1044
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1045
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1046
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1047
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1048
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1049
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1050
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1051
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1052
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1053
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1054
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1055
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1056
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1057
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1058
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1059
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1060
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1061
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1062
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1063
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1064
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1065
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1066
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1067
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1068
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1069
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1070
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1071
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1072
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1073
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1074
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1075
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1076
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1077
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1078
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1079
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1080
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1081
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1082
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1083
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1084
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1085
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1086
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1087
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1088
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1089
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1090
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1091
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1092
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1093
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1094
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1095
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1096
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1097
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1098
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1099
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1100
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1101
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1102
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1103
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1104
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1105
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1106
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1107
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1108
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1109
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1110
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1111
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1112
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1113
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1114
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1115
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1116
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1117
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1118
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1119
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1120
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1121
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1122
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1123
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1124
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1125
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1126
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1127
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1128
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1129
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1130
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1131
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1132
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1133
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1134
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1135
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1136
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1137
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1138
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1139
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1140
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1141
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1142
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1143
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1144
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1145
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1146
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1147
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1148
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1149
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1150
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1151
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1152
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1153
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1154
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1155
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1156
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1157
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1158
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1159
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1160
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1161
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1162
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1163
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1164
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1165
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1166
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1167
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1168
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1169
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1170
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1171
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1172
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1173
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1174
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1175
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1176
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1177
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1178
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1179
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1180
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1181
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1182
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1183
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1184
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1185
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1186
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1187
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1188
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1189
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1190
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1191
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1192
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1193
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1194
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1195
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1196
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1197
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1198
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1199
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1200
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1201
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1202
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1203
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1204
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1205
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1206
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1207
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1208
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1209
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1210
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1211
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1212
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1213
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1214
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1215
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1216
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1217
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1218
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1219
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1220
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1221
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1222
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1223
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1224
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1225
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1226
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1227
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1228
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1229
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1230
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1231
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1232
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1233
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1234
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1235
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1236
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1237
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1238
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1239
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1240
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1241
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1242
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1243
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1244
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1245
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1246
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1247
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1248
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1249
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1250
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1251
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1252
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1253
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1254
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1255
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1256
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1257
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1258
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1259
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1260
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1261
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1262
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1263
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1264
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1265
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1266
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1267
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1268
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1269
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1270
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1271
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1272
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1273
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1274
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1275
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1276
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1277
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1278
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1279
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher4()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel4(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 1024: LAUNCH_KERNEL(1024); break;
	case 1025: LAUNCH_KERNEL(1025); break;
	case 1026: LAUNCH_KERNEL(1026); break;
	case 1027: LAUNCH_KERNEL(1027); break;
	case 1028: LAUNCH_KERNEL(1028); break;
	case 1029: LAUNCH_KERNEL(1029); break;
	case 1030: LAUNCH_KERNEL(1030); break;
	case 1031: LAUNCH_KERNEL(1031); break;
	case 1032: LAUNCH_KERNEL(1032); break;
	case 1033: LAUNCH_KERNEL(1033); break;
	case 1034: LAUNCH_KERNEL(1034); break;
	case 1035: LAUNCH_KERNEL(1035); break;
	case 1036: LAUNCH_KERNEL(1036); break;
	case 1037: LAUNCH_KERNEL(1037); break;
	case 1038: LAUNCH_KERNEL(1038); break;
	case 1039: LAUNCH_KERNEL(1039); break;
	case 1040: LAUNCH_KERNEL(1040); break;
	case 1041: LAUNCH_KERNEL(1041); break;
	case 1042: LAUNCH_KERNEL(1042); break;
	case 1043: LAUNCH_KERNEL(1043); break;
	case 1044: LAUNCH_KERNEL(1044); break;
	case 1045: LAUNCH_KERNEL(1045); break;
	case 1046: LAUNCH_KERNEL(1046); break;
	case 1047: LAUNCH_KERNEL(1047); break;
	case 1048: LAUNCH_KERNEL(1048); break;
	case 1049: LAUNCH_KERNEL(1049); break;
	case 1050: LAUNCH_KERNEL(1050); break;
	case 1051: LAUNCH_KERNEL(1051); break;
	case 1052: LAUNCH_KERNEL(1052); break;
	case 1053: LAUNCH_KERNEL(1053); break;
	case 1054: LAUNCH_KERNEL(1054); break;
	case 1055: LAUNCH_KERNEL(1055); break;
	case 1056: LAUNCH_KERNEL(1056); break;
	case 1057: LAUNCH_KERNEL(1057); break;
	case 1058: LAUNCH_KERNEL(1058); break;
	case 1059: LAUNCH_KERNEL(1059); break;
	case 1060: LAUNCH_KERNEL(1060); break;
	case 1061: LAUNCH_KERNEL(1061); break;
	case 1062: LAUNCH_KERNEL(1062); break;
	case 1063: LAUNCH_KERNEL(1063); break;
	case 1064: LAUNCH_KERNEL(1064); break;
	case 1065: LAUNCH_KERNEL(1065); break;
	case 1066: LAUNCH_KERNEL(1066); break;
	case 1067: LAUNCH_KERNEL(1067); break;
	case 1068: LAUNCH_KERNEL(1068); break;
	case 1069: LAUNCH_KERNEL(1069); break;
	case 1070: LAUNCH_KERNEL(1070); break;
	case 1071: LAUNCH_KERNEL(1071); break;
	case 1072: LAUNCH_KERNEL(1072); break;
	case 1073: LAUNCH_KERNEL(1073); break;
	case 1074: LAUNCH_KERNEL(1074); break;
	case 1075: LAUNCH_KERNEL(1075); break;
	case 1076: LAUNCH_KERNEL(1076); break;
	case 1077: LAUNCH_KERNEL(1077); break;
	case 1078: LAUNCH_KERNEL(1078); break;
	case 1079: LAUNCH_KERNEL(1079); break;
	case 1080: LAUNCH_KERNEL(1080); break;
	case 1081: LAUNCH_KERNEL(1081); break;
	case 1082: LAUNCH_KERNEL(1082); break;
	case 1083: LAUNCH_KERNEL(1083); break;
	case 1084: LAUNCH_KERNEL(1084); break;
	case 1085: LAUNCH_KERNEL(1085); break;
	case 1086: LAUNCH_KERNEL(1086); break;
	case 1087: LAUNCH_KERNEL(1087); break;
	case 1088: LAUNCH_KERNEL(1088); break;
	case 1089: LAUNCH_KERNEL(1089); break;
	case 1090: LAUNCH_KERNEL(1090); break;
	case 1091: LAUNCH_KERNEL(1091); break;
	case 1092: LAUNCH_KERNEL(1092); break;
	case 1093: LAUNCH_KERNEL(1093); break;
	case 1094: LAUNCH_KERNEL(1094); break;
	case 1095: LAUNCH_KERNEL(1095); break;
	case 1096: LAUNCH_KERNEL(1096); break;
	case 1097: LAUNCH_KERNEL(1097); break;
	case 1098: LAUNCH_KERNEL(1098); break;
	case 1099: LAUNCH_KERNEL(1099); break;
	case 1100: LAUNCH_KERNEL(1100); break;
	case 1101: LAUNCH_KERNEL(1101); break;
	case 1102: LAUNCH_KERNEL(1102); break;
	case 1103: LAUNCH_KERNEL(1103); break;
	case 1104: LAUNCH_KERNEL(1104); break;
	case 1105: LAUNCH_KERNEL(1105); break;
	case 1106: LAUNCH_KERNEL(1106); break;
	case 1107: LAUNCH_KERNEL(1107); break;
	case 1108: LAUNCH_KERNEL(1108); break;
	case 1109: LAUNCH_KERNEL(1109); break;
	case 1110: LAUNCH_KERNEL(1110); break;
	case 1111: LAUNCH_KERNEL(1111); break;
	case 1112: LAUNCH_KERNEL(1112); break;
	case 1113: LAUNCH_KERNEL(1113); break;
	case 1114: LAUNCH_KERNEL(1114); break;
	case 1115: LAUNCH_KERNEL(1115); break;
	case 1116: LAUNCH_KERNEL(1116); break;
	case 1117: LAUNCH_KERNEL(1117); break;
	case 1118: LAUNCH_KERNEL(1118); break;
	case 1119: LAUNCH_KERNEL(1119); break;
	case 1120: LAUNCH_KERNEL(1120); break;
	case 1121: LAUNCH_KERNEL(1121); break;
	case 1122: LAUNCH_KERNEL(1122); break;
	case 1123: LAUNCH_KERNEL(1123); break;
	case 1124: LAUNCH_KERNEL(1124); break;
	case 1125: LAUNCH_KERNEL(1125); break;
	case 1126: LAUNCH_KERNEL(1126); break;
	case 1127: LAUNCH_KERNEL(1127); break;
	case 1128: LAUNCH_KERNEL(1128); break;
	case 1129: LAUNCH_KERNEL(1129); break;
	case 1130: LAUNCH_KERNEL(1130); break;
	case 1131: LAUNCH_KERNEL(1131); break;
	case 1132: LAUNCH_KERNEL(1132); break;
	case 1133: LAUNCH_KERNEL(1133); break;
	case 1134: LAUNCH_KERNEL(1134); break;
	case 1135: LAUNCH_KERNEL(1135); break;
	case 1136: LAUNCH_KERNEL(1136); break;
	case 1137: LAUNCH_KERNEL(1137); break;
	case 1138: LAUNCH_KERNEL(1138); break;
	case 1139: LAUNCH_KERNEL(1139); break;
	case 1140: LAUNCH_KERNEL(1140); break;
	case 1141: LAUNCH_KERNEL(1141); break;
	case 1142: LAUNCH_KERNEL(1142); break;
	case 1143: LAUNCH_KERNEL(1143); break;
	case 1144: LAUNCH_KERNEL(1144); break;
	case 1145: LAUNCH_KERNEL(1145); break;
	case 1146: LAUNCH_KERNEL(1146); break;
	case 1147: LAUNCH_KERNEL(1147); break;
	case 1148: LAUNCH_KERNEL(1148); break;
	case 1149: LAUNCH_KERNEL(1149); break;
	case 1150: LAUNCH_KERNEL(1150); break;
	case 1151: LAUNCH_KERNEL(1151); break;
	case 1152: LAUNCH_KERNEL(1152); break;
	case 1153: LAUNCH_KERNEL(1153); break;
	case 1154: LAUNCH_KERNEL(1154); break;
	case 1155: LAUNCH_KERNEL(1155); break;
	case 1156: LAUNCH_KERNEL(1156); break;
	case 1157: LAUNCH_KERNEL(1157); break;
	case 1158: LAUNCH_KERNEL(1158); break;
	case 1159: LAUNCH_KERNEL(1159); break;
	case 1160: LAUNCH_KERNEL(1160); break;
	case 1161: LAUNCH_KERNEL(1161); break;
	case 1162: LAUNCH_KERNEL(1162); break;
	case 1163: LAUNCH_KERNEL(1163); break;
	case 1164: LAUNCH_KERNEL(1164); break;
	case 1165: LAUNCH_KERNEL(1165); break;
	case 1166: LAUNCH_KERNEL(1166); break;
	case 1167: LAUNCH_KERNEL(1167); break;
	case 1168: LAUNCH_KERNEL(1168); break;
	case 1169: LAUNCH_KERNEL(1169); break;
	case 1170: LAUNCH_KERNEL(1170); break;
	case 1171: LAUNCH_KERNEL(1171); break;
	case 1172: LAUNCH_KERNEL(1172); break;
	case 1173: LAUNCH_KERNEL(1173); break;
	case 1174: LAUNCH_KERNEL(1174); break;
	case 1175: LAUNCH_KERNEL(1175); break;
	case 1176: LAUNCH_KERNEL(1176); break;
	case 1177: LAUNCH_KERNEL(1177); break;
	case 1178: LAUNCH_KERNEL(1178); break;
	case 1179: LAUNCH_KERNEL(1179); break;
	case 1180: LAUNCH_KERNEL(1180); break;
	case 1181: LAUNCH_KERNEL(1181); break;
	case 1182: LAUNCH_KERNEL(1182); break;
	case 1183: LAUNCH_KERNEL(1183); break;
	case 1184: LAUNCH_KERNEL(1184); break;
	case 1185: LAUNCH_KERNEL(1185); break;
	case 1186: LAUNCH_KERNEL(1186); break;
	case 1187: LAUNCH_KERNEL(1187); break;
	case 1188: LAUNCH_KERNEL(1188); break;
	case 1189: LAUNCH_KERNEL(1189); break;
	case 1190: LAUNCH_KERNEL(1190); break;
	case 1191: LAUNCH_KERNEL(1191); break;
	case 1192: LAUNCH_KERNEL(1192); break;
	case 1193: LAUNCH_KERNEL(1193); break;
	case 1194: LAUNCH_KERNEL(1194); break;
	case 1195: LAUNCH_KERNEL(1195); break;
	case 1196: LAUNCH_KERNEL(1196); break;
	case 1197: LAUNCH_KERNEL(1197); break;
	case 1198: LAUNCH_KERNEL(1198); break;
	case 1199: LAUNCH_KERNEL(1199); break;
	case 1200: LAUNCH_KERNEL(1200); break;
	case 1201: LAUNCH_KERNEL(1201); break;
	case 1202: LAUNCH_KERNEL(1202); break;
	case 1203: LAUNCH_KERNEL(1203); break;
	case 1204: LAUNCH_KERNEL(1204); break;
	case 1205: LAUNCH_KERNEL(1205); break;
	case 1206: LAUNCH_KERNEL(1206); break;
	case 1207: LAUNCH_KERNEL(1207); break;
	case 1208: LAUNCH_KERNEL(1208); break;
	case 1209: LAUNCH_KERNEL(1209); break;
	case 1210: LAUNCH_KERNEL(1210); break;
	case 1211: LAUNCH_KERNEL(1211); break;
	case 1212: LAUNCH_KERNEL(1212); break;
	case 1213: LAUNCH_KERNEL(1213); break;
	case 1214: LAUNCH_KERNEL(1214); break;
	case 1215: LAUNCH_KERNEL(1215); break;
	case 1216: LAUNCH_KERNEL(1216); break;
	case 1217: LAUNCH_KERNEL(1217); break;
	case 1218: LAUNCH_KERNEL(1218); break;
	case 1219: LAUNCH_KERNEL(1219); break;
	case 1220: LAUNCH_KERNEL(1220); break;
	case 1221: LAUNCH_KERNEL(1221); break;
	case 1222: LAUNCH_KERNEL(1222); break;
	case 1223: LAUNCH_KERNEL(1223); break;
	case 1224: LAUNCH_KERNEL(1224); break;
	case 1225: LAUNCH_KERNEL(1225); break;
	case 1226: LAUNCH_KERNEL(1226); break;
	case 1227: LAUNCH_KERNEL(1227); break;
	case 1228: LAUNCH_KERNEL(1228); break;
	case 1229: LAUNCH_KERNEL(1229); break;
	case 1230: LAUNCH_KERNEL(1230); break;
	case 1231: LAUNCH_KERNEL(1231); break;
	case 1232: LAUNCH_KERNEL(1232); break;
	case 1233: LAUNCH_KERNEL(1233); break;
	case 1234: LAUNCH_KERNEL(1234); break;
	case 1235: LAUNCH_KERNEL(1235); break;
	case 1236: LAUNCH_KERNEL(1236); break;
	case 1237: LAUNCH_KERNEL(1237); break;
	case 1238: LAUNCH_KERNEL(1238); break;
	case 1239: LAUNCH_KERNEL(1239); break;
	case 1240: LAUNCH_KERNEL(1240); break;
	case 1241: LAUNCH_KERNEL(1241); break;
	case 1242: LAUNCH_KERNEL(1242); break;
	case 1243: LAUNCH_KERNEL(1243); break;
	case 1244: LAUNCH_KERNEL(1244); break;
	case 1245: LAUNCH_KERNEL(1245); break;
	case 1246: LAUNCH_KERNEL(1246); break;
	case 1247: LAUNCH_KERNEL(1247); break;
	case 1248: LAUNCH_KERNEL(1248); break;
	case 1249: LAUNCH_KERNEL(1249); break;
	case 1250: LAUNCH_KERNEL(1250); break;
	case 1251: LAUNCH_KERNEL(1251); break;
	case 1252: LAUNCH_KERNEL(1252); break;
	case 1253: LAUNCH_KERNEL(1253); break;
	case 1254: LAUNCH_KERNEL(1254); break;
	case 1255: LAUNCH_KERNEL(1255); break;
	case 1256: LAUNCH_KERNEL(1256); break;
	case 1257: LAUNCH_KERNEL(1257); break;
	case 1258: LAUNCH_KERNEL(1258); break;
	case 1259: LAUNCH_KERNEL(1259); break;
	case 1260: LAUNCH_KERNEL(1260); break;
	case 1261: LAUNCH_KERNEL(1261); break;
	case 1262: LAUNCH_KERNEL(1262); break;
	case 1263: LAUNCH_KERNEL(1263); break;
	case 1264: LAUNCH_KERNEL(1264); break;
	case 1265: LAUNCH_KERNEL(1265); break;
	case 1266: LAUNCH_KERNEL(1266); break;
	case 1267: LAUNCH_KERNEL(1267); break;
	case 1268: LAUNCH_KERNEL(1268); break;
	case 1269: LAUNCH_KERNEL(1269); break;
	case 1270: LAUNCH_KERNEL(1270); break;
	case 1271: LAUNCH_KERNEL(1271); break;
	case 1272: LAUNCH_KERNEL(1272); break;
	case 1273: LAUNCH_KERNEL(1273); break;
	case 1274: LAUNCH_KERNEL(1274); break;
	case 1275: LAUNCH_KERNEL(1275); break;
	case 1276: LAUNCH_KERNEL(1276); break;
	case 1277: LAUNCH_KERNEL(1277); break;
	case 1278: LAUNCH_KERNEL(1278); break;
	case 1279: LAUNCH_KERNEL(1279); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
