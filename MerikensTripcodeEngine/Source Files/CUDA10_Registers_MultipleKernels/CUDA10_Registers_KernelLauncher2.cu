// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 512
#include "../CUDA10_Registers_Kernel.h"
#define SALT 513
#include "../CUDA10_Registers_Kernel.h"
#define SALT 514
#include "../CUDA10_Registers_Kernel.h"
#define SALT 515
#include "../CUDA10_Registers_Kernel.h"
#define SALT 516
#include "../CUDA10_Registers_Kernel.h"
#define SALT 517
#include "../CUDA10_Registers_Kernel.h"
#define SALT 518
#include "../CUDA10_Registers_Kernel.h"
#define SALT 519
#include "../CUDA10_Registers_Kernel.h"
#define SALT 520
#include "../CUDA10_Registers_Kernel.h"
#define SALT 521
#include "../CUDA10_Registers_Kernel.h"
#define SALT 522
#include "../CUDA10_Registers_Kernel.h"
#define SALT 523
#include "../CUDA10_Registers_Kernel.h"
#define SALT 524
#include "../CUDA10_Registers_Kernel.h"
#define SALT 525
#include "../CUDA10_Registers_Kernel.h"
#define SALT 526
#include "../CUDA10_Registers_Kernel.h"
#define SALT 527
#include "../CUDA10_Registers_Kernel.h"
#define SALT 528
#include "../CUDA10_Registers_Kernel.h"
#define SALT 529
#include "../CUDA10_Registers_Kernel.h"
#define SALT 530
#include "../CUDA10_Registers_Kernel.h"
#define SALT 531
#include "../CUDA10_Registers_Kernel.h"
#define SALT 532
#include "../CUDA10_Registers_Kernel.h"
#define SALT 533
#include "../CUDA10_Registers_Kernel.h"
#define SALT 534
#include "../CUDA10_Registers_Kernel.h"
#define SALT 535
#include "../CUDA10_Registers_Kernel.h"
#define SALT 536
#include "../CUDA10_Registers_Kernel.h"
#define SALT 537
#include "../CUDA10_Registers_Kernel.h"
#define SALT 538
#include "../CUDA10_Registers_Kernel.h"
#define SALT 539
#include "../CUDA10_Registers_Kernel.h"
#define SALT 540
#include "../CUDA10_Registers_Kernel.h"
#define SALT 541
#include "../CUDA10_Registers_Kernel.h"
#define SALT 542
#include "../CUDA10_Registers_Kernel.h"
#define SALT 543
#include "../CUDA10_Registers_Kernel.h"
#define SALT 544
#include "../CUDA10_Registers_Kernel.h"
#define SALT 545
#include "../CUDA10_Registers_Kernel.h"
#define SALT 546
#include "../CUDA10_Registers_Kernel.h"
#define SALT 547
#include "../CUDA10_Registers_Kernel.h"
#define SALT 548
#include "../CUDA10_Registers_Kernel.h"
#define SALT 549
#include "../CUDA10_Registers_Kernel.h"
#define SALT 550
#include "../CUDA10_Registers_Kernel.h"
#define SALT 551
#include "../CUDA10_Registers_Kernel.h"
#define SALT 552
#include "../CUDA10_Registers_Kernel.h"
#define SALT 553
#include "../CUDA10_Registers_Kernel.h"
#define SALT 554
#include "../CUDA10_Registers_Kernel.h"
#define SALT 555
#include "../CUDA10_Registers_Kernel.h"
#define SALT 556
#include "../CUDA10_Registers_Kernel.h"
#define SALT 557
#include "../CUDA10_Registers_Kernel.h"
#define SALT 558
#include "../CUDA10_Registers_Kernel.h"
#define SALT 559
#include "../CUDA10_Registers_Kernel.h"
#define SALT 560
#include "../CUDA10_Registers_Kernel.h"
#define SALT 561
#include "../CUDA10_Registers_Kernel.h"
#define SALT 562
#include "../CUDA10_Registers_Kernel.h"
#define SALT 563
#include "../CUDA10_Registers_Kernel.h"
#define SALT 564
#include "../CUDA10_Registers_Kernel.h"
#define SALT 565
#include "../CUDA10_Registers_Kernel.h"
#define SALT 566
#include "../CUDA10_Registers_Kernel.h"
#define SALT 567
#include "../CUDA10_Registers_Kernel.h"
#define SALT 568
#include "../CUDA10_Registers_Kernel.h"
#define SALT 569
#include "../CUDA10_Registers_Kernel.h"
#define SALT 570
#include "../CUDA10_Registers_Kernel.h"
#define SALT 571
#include "../CUDA10_Registers_Kernel.h"
#define SALT 572
#include "../CUDA10_Registers_Kernel.h"
#define SALT 573
#include "../CUDA10_Registers_Kernel.h"
#define SALT 574
#include "../CUDA10_Registers_Kernel.h"
#define SALT 575
#include "../CUDA10_Registers_Kernel.h"
#define SALT 576
#include "../CUDA10_Registers_Kernel.h"
#define SALT 577
#include "../CUDA10_Registers_Kernel.h"
#define SALT 578
#include "../CUDA10_Registers_Kernel.h"
#define SALT 579
#include "../CUDA10_Registers_Kernel.h"
#define SALT 580
#include "../CUDA10_Registers_Kernel.h"
#define SALT 581
#include "../CUDA10_Registers_Kernel.h"
#define SALT 582
#include "../CUDA10_Registers_Kernel.h"
#define SALT 583
#include "../CUDA10_Registers_Kernel.h"
#define SALT 584
#include "../CUDA10_Registers_Kernel.h"
#define SALT 585
#include "../CUDA10_Registers_Kernel.h"
#define SALT 586
#include "../CUDA10_Registers_Kernel.h"
#define SALT 587
#include "../CUDA10_Registers_Kernel.h"
#define SALT 588
#include "../CUDA10_Registers_Kernel.h"
#define SALT 589
#include "../CUDA10_Registers_Kernel.h"
#define SALT 590
#include "../CUDA10_Registers_Kernel.h"
#define SALT 591
#include "../CUDA10_Registers_Kernel.h"
#define SALT 592
#include "../CUDA10_Registers_Kernel.h"
#define SALT 593
#include "../CUDA10_Registers_Kernel.h"
#define SALT 594
#include "../CUDA10_Registers_Kernel.h"
#define SALT 595
#include "../CUDA10_Registers_Kernel.h"
#define SALT 596
#include "../CUDA10_Registers_Kernel.h"
#define SALT 597
#include "../CUDA10_Registers_Kernel.h"
#define SALT 598
#include "../CUDA10_Registers_Kernel.h"
#define SALT 599
#include "../CUDA10_Registers_Kernel.h"
#define SALT 600
#include "../CUDA10_Registers_Kernel.h"
#define SALT 601
#include "../CUDA10_Registers_Kernel.h"
#define SALT 602
#include "../CUDA10_Registers_Kernel.h"
#define SALT 603
#include "../CUDA10_Registers_Kernel.h"
#define SALT 604
#include "../CUDA10_Registers_Kernel.h"
#define SALT 605
#include "../CUDA10_Registers_Kernel.h"
#define SALT 606
#include "../CUDA10_Registers_Kernel.h"
#define SALT 607
#include "../CUDA10_Registers_Kernel.h"
#define SALT 608
#include "../CUDA10_Registers_Kernel.h"
#define SALT 609
#include "../CUDA10_Registers_Kernel.h"
#define SALT 610
#include "../CUDA10_Registers_Kernel.h"
#define SALT 611
#include "../CUDA10_Registers_Kernel.h"
#define SALT 612
#include "../CUDA10_Registers_Kernel.h"
#define SALT 613
#include "../CUDA10_Registers_Kernel.h"
#define SALT 614
#include "../CUDA10_Registers_Kernel.h"
#define SALT 615
#include "../CUDA10_Registers_Kernel.h"
#define SALT 616
#include "../CUDA10_Registers_Kernel.h"
#define SALT 617
#include "../CUDA10_Registers_Kernel.h"
#define SALT 618
#include "../CUDA10_Registers_Kernel.h"
#define SALT 619
#include "../CUDA10_Registers_Kernel.h"
#define SALT 620
#include "../CUDA10_Registers_Kernel.h"
#define SALT 621
#include "../CUDA10_Registers_Kernel.h"
#define SALT 622
#include "../CUDA10_Registers_Kernel.h"
#define SALT 623
#include "../CUDA10_Registers_Kernel.h"
#define SALT 624
#include "../CUDA10_Registers_Kernel.h"
#define SALT 625
#include "../CUDA10_Registers_Kernel.h"
#define SALT 626
#include "../CUDA10_Registers_Kernel.h"
#define SALT 627
#include "../CUDA10_Registers_Kernel.h"
#define SALT 628
#include "../CUDA10_Registers_Kernel.h"
#define SALT 629
#include "../CUDA10_Registers_Kernel.h"
#define SALT 630
#include "../CUDA10_Registers_Kernel.h"
#define SALT 631
#include "../CUDA10_Registers_Kernel.h"
#define SALT 632
#include "../CUDA10_Registers_Kernel.h"
#define SALT 633
#include "../CUDA10_Registers_Kernel.h"
#define SALT 634
#include "../CUDA10_Registers_Kernel.h"
#define SALT 635
#include "../CUDA10_Registers_Kernel.h"
#define SALT 636
#include "../CUDA10_Registers_Kernel.h"
#define SALT 637
#include "../CUDA10_Registers_Kernel.h"
#define SALT 638
#include "../CUDA10_Registers_Kernel.h"
#define SALT 639
#include "../CUDA10_Registers_Kernel.h"
#define SALT 640
#include "../CUDA10_Registers_Kernel.h"
#define SALT 641
#include "../CUDA10_Registers_Kernel.h"
#define SALT 642
#include "../CUDA10_Registers_Kernel.h"
#define SALT 643
#include "../CUDA10_Registers_Kernel.h"
#define SALT 644
#include "../CUDA10_Registers_Kernel.h"
#define SALT 645
#include "../CUDA10_Registers_Kernel.h"
#define SALT 646
#include "../CUDA10_Registers_Kernel.h"
#define SALT 647
#include "../CUDA10_Registers_Kernel.h"
#define SALT 648
#include "../CUDA10_Registers_Kernel.h"
#define SALT 649
#include "../CUDA10_Registers_Kernel.h"
#define SALT 650
#include "../CUDA10_Registers_Kernel.h"
#define SALT 651
#include "../CUDA10_Registers_Kernel.h"
#define SALT 652
#include "../CUDA10_Registers_Kernel.h"
#define SALT 653
#include "../CUDA10_Registers_Kernel.h"
#define SALT 654
#include "../CUDA10_Registers_Kernel.h"
#define SALT 655
#include "../CUDA10_Registers_Kernel.h"
#define SALT 656
#include "../CUDA10_Registers_Kernel.h"
#define SALT 657
#include "../CUDA10_Registers_Kernel.h"
#define SALT 658
#include "../CUDA10_Registers_Kernel.h"
#define SALT 659
#include "../CUDA10_Registers_Kernel.h"
#define SALT 660
#include "../CUDA10_Registers_Kernel.h"
#define SALT 661
#include "../CUDA10_Registers_Kernel.h"
#define SALT 662
#include "../CUDA10_Registers_Kernel.h"
#define SALT 663
#include "../CUDA10_Registers_Kernel.h"
#define SALT 664
#include "../CUDA10_Registers_Kernel.h"
#define SALT 665
#include "../CUDA10_Registers_Kernel.h"
#define SALT 666
#include "../CUDA10_Registers_Kernel.h"
#define SALT 667
#include "../CUDA10_Registers_Kernel.h"
#define SALT 668
#include "../CUDA10_Registers_Kernel.h"
#define SALT 669
#include "../CUDA10_Registers_Kernel.h"
#define SALT 670
#include "../CUDA10_Registers_Kernel.h"
#define SALT 671
#include "../CUDA10_Registers_Kernel.h"
#define SALT 672
#include "../CUDA10_Registers_Kernel.h"
#define SALT 673
#include "../CUDA10_Registers_Kernel.h"
#define SALT 674
#include "../CUDA10_Registers_Kernel.h"
#define SALT 675
#include "../CUDA10_Registers_Kernel.h"
#define SALT 676
#include "../CUDA10_Registers_Kernel.h"
#define SALT 677
#include "../CUDA10_Registers_Kernel.h"
#define SALT 678
#include "../CUDA10_Registers_Kernel.h"
#define SALT 679
#include "../CUDA10_Registers_Kernel.h"
#define SALT 680
#include "../CUDA10_Registers_Kernel.h"
#define SALT 681
#include "../CUDA10_Registers_Kernel.h"
#define SALT 682
#include "../CUDA10_Registers_Kernel.h"
#define SALT 683
#include "../CUDA10_Registers_Kernel.h"
#define SALT 684
#include "../CUDA10_Registers_Kernel.h"
#define SALT 685
#include "../CUDA10_Registers_Kernel.h"
#define SALT 686
#include "../CUDA10_Registers_Kernel.h"
#define SALT 687
#include "../CUDA10_Registers_Kernel.h"
#define SALT 688
#include "../CUDA10_Registers_Kernel.h"
#define SALT 689
#include "../CUDA10_Registers_Kernel.h"
#define SALT 690
#include "../CUDA10_Registers_Kernel.h"
#define SALT 691
#include "../CUDA10_Registers_Kernel.h"
#define SALT 692
#include "../CUDA10_Registers_Kernel.h"
#define SALT 693
#include "../CUDA10_Registers_Kernel.h"
#define SALT 694
#include "../CUDA10_Registers_Kernel.h"
#define SALT 695
#include "../CUDA10_Registers_Kernel.h"
#define SALT 696
#include "../CUDA10_Registers_Kernel.h"
#define SALT 697
#include "../CUDA10_Registers_Kernel.h"
#define SALT 698
#include "../CUDA10_Registers_Kernel.h"
#define SALT 699
#include "../CUDA10_Registers_Kernel.h"
#define SALT 700
#include "../CUDA10_Registers_Kernel.h"
#define SALT 701
#include "../CUDA10_Registers_Kernel.h"
#define SALT 702
#include "../CUDA10_Registers_Kernel.h"
#define SALT 703
#include "../CUDA10_Registers_Kernel.h"
#define SALT 704
#include "../CUDA10_Registers_Kernel.h"
#define SALT 705
#include "../CUDA10_Registers_Kernel.h"
#define SALT 706
#include "../CUDA10_Registers_Kernel.h"
#define SALT 707
#include "../CUDA10_Registers_Kernel.h"
#define SALT 708
#include "../CUDA10_Registers_Kernel.h"
#define SALT 709
#include "../CUDA10_Registers_Kernel.h"
#define SALT 710
#include "../CUDA10_Registers_Kernel.h"
#define SALT 711
#include "../CUDA10_Registers_Kernel.h"
#define SALT 712
#include "../CUDA10_Registers_Kernel.h"
#define SALT 713
#include "../CUDA10_Registers_Kernel.h"
#define SALT 714
#include "../CUDA10_Registers_Kernel.h"
#define SALT 715
#include "../CUDA10_Registers_Kernel.h"
#define SALT 716
#include "../CUDA10_Registers_Kernel.h"
#define SALT 717
#include "../CUDA10_Registers_Kernel.h"
#define SALT 718
#include "../CUDA10_Registers_Kernel.h"
#define SALT 719
#include "../CUDA10_Registers_Kernel.h"
#define SALT 720
#include "../CUDA10_Registers_Kernel.h"
#define SALT 721
#include "../CUDA10_Registers_Kernel.h"
#define SALT 722
#include "../CUDA10_Registers_Kernel.h"
#define SALT 723
#include "../CUDA10_Registers_Kernel.h"
#define SALT 724
#include "../CUDA10_Registers_Kernel.h"
#define SALT 725
#include "../CUDA10_Registers_Kernel.h"
#define SALT 726
#include "../CUDA10_Registers_Kernel.h"
#define SALT 727
#include "../CUDA10_Registers_Kernel.h"
#define SALT 728
#include "../CUDA10_Registers_Kernel.h"
#define SALT 729
#include "../CUDA10_Registers_Kernel.h"
#define SALT 730
#include "../CUDA10_Registers_Kernel.h"
#define SALT 731
#include "../CUDA10_Registers_Kernel.h"
#define SALT 732
#include "../CUDA10_Registers_Kernel.h"
#define SALT 733
#include "../CUDA10_Registers_Kernel.h"
#define SALT 734
#include "../CUDA10_Registers_Kernel.h"
#define SALT 735
#include "../CUDA10_Registers_Kernel.h"
#define SALT 736
#include "../CUDA10_Registers_Kernel.h"
#define SALT 737
#include "../CUDA10_Registers_Kernel.h"
#define SALT 738
#include "../CUDA10_Registers_Kernel.h"
#define SALT 739
#include "../CUDA10_Registers_Kernel.h"
#define SALT 740
#include "../CUDA10_Registers_Kernel.h"
#define SALT 741
#include "../CUDA10_Registers_Kernel.h"
#define SALT 742
#include "../CUDA10_Registers_Kernel.h"
#define SALT 743
#include "../CUDA10_Registers_Kernel.h"
#define SALT 744
#include "../CUDA10_Registers_Kernel.h"
#define SALT 745
#include "../CUDA10_Registers_Kernel.h"
#define SALT 746
#include "../CUDA10_Registers_Kernel.h"
#define SALT 747
#include "../CUDA10_Registers_Kernel.h"
#define SALT 748
#include "../CUDA10_Registers_Kernel.h"
#define SALT 749
#include "../CUDA10_Registers_Kernel.h"
#define SALT 750
#include "../CUDA10_Registers_Kernel.h"
#define SALT 751
#include "../CUDA10_Registers_Kernel.h"
#define SALT 752
#include "../CUDA10_Registers_Kernel.h"
#define SALT 753
#include "../CUDA10_Registers_Kernel.h"
#define SALT 754
#include "../CUDA10_Registers_Kernel.h"
#define SALT 755
#include "../CUDA10_Registers_Kernel.h"
#define SALT 756
#include "../CUDA10_Registers_Kernel.h"
#define SALT 757
#include "../CUDA10_Registers_Kernel.h"
#define SALT 758
#include "../CUDA10_Registers_Kernel.h"
#define SALT 759
#include "../CUDA10_Registers_Kernel.h"
#define SALT 760
#include "../CUDA10_Registers_Kernel.h"
#define SALT 761
#include "../CUDA10_Registers_Kernel.h"
#define SALT 762
#include "../CUDA10_Registers_Kernel.h"
#define SALT 763
#include "../CUDA10_Registers_Kernel.h"
#define SALT 764
#include "../CUDA10_Registers_Kernel.h"
#define SALT 765
#include "../CUDA10_Registers_Kernel.h"
#define SALT 766
#include "../CUDA10_Registers_Kernel.h"
#define SALT 767
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher2()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel2(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
		case 512: LAUNCH_KERNEL(512); break;
		case 513: LAUNCH_KERNEL(513); break;
		case 514: LAUNCH_KERNEL(514); break;
		case 515: LAUNCH_KERNEL(515); break;
		case 516: LAUNCH_KERNEL(516); break;
		case 517: LAUNCH_KERNEL(517); break;
		case 518: LAUNCH_KERNEL(518); break;
		case 519: LAUNCH_KERNEL(519); break;
		case 520: LAUNCH_KERNEL(520); break;
		case 521: LAUNCH_KERNEL(521); break;
		case 522: LAUNCH_KERNEL(522); break;
		case 523: LAUNCH_KERNEL(523); break;
		case 524: LAUNCH_KERNEL(524); break;
		case 525: LAUNCH_KERNEL(525); break;
		case 526: LAUNCH_KERNEL(526); break;
		case 527: LAUNCH_KERNEL(527); break;
		case 528: LAUNCH_KERNEL(528); break;
		case 529: LAUNCH_KERNEL(529); break;
		case 530: LAUNCH_KERNEL(530); break;
		case 531: LAUNCH_KERNEL(531); break;
		case 532: LAUNCH_KERNEL(532); break;
		case 533: LAUNCH_KERNEL(533); break;
		case 534: LAUNCH_KERNEL(534); break;
		case 535: LAUNCH_KERNEL(535); break;
		case 536: LAUNCH_KERNEL(536); break;
		case 537: LAUNCH_KERNEL(537); break;
		case 538: LAUNCH_KERNEL(538); break;
		case 539: LAUNCH_KERNEL(539); break;
		case 540: LAUNCH_KERNEL(540); break;
		case 541: LAUNCH_KERNEL(541); break;
		case 542: LAUNCH_KERNEL(542); break;
		case 543: LAUNCH_KERNEL(543); break;
		case 544: LAUNCH_KERNEL(544); break;
		case 545: LAUNCH_KERNEL(545); break;
		case 546: LAUNCH_KERNEL(546); break;
		case 547: LAUNCH_KERNEL(547); break;
		case 548: LAUNCH_KERNEL(548); break;
		case 549: LAUNCH_KERNEL(549); break;
		case 550: LAUNCH_KERNEL(550); break;
		case 551: LAUNCH_KERNEL(551); break;
		case 552: LAUNCH_KERNEL(552); break;
		case 553: LAUNCH_KERNEL(553); break;
		case 554: LAUNCH_KERNEL(554); break;
		case 555: LAUNCH_KERNEL(555); break;
		case 556: LAUNCH_KERNEL(556); break;
		case 557: LAUNCH_KERNEL(557); break;
		case 558: LAUNCH_KERNEL(558); break;
		case 559: LAUNCH_KERNEL(559); break;
		case 560: LAUNCH_KERNEL(560); break;
		case 561: LAUNCH_KERNEL(561); break;
		case 562: LAUNCH_KERNEL(562); break;
		case 563: LAUNCH_KERNEL(563); break;
		case 564: LAUNCH_KERNEL(564); break;
		case 565: LAUNCH_KERNEL(565); break;
		case 566: LAUNCH_KERNEL(566); break;
		case 567: LAUNCH_KERNEL(567); break;
		case 568: LAUNCH_KERNEL(568); break;
		case 569: LAUNCH_KERNEL(569); break;
		case 570: LAUNCH_KERNEL(570); break;
		case 571: LAUNCH_KERNEL(571); break;
		case 572: LAUNCH_KERNEL(572); break;
		case 573: LAUNCH_KERNEL(573); break;
		case 574: LAUNCH_KERNEL(574); break;
		case 575: LAUNCH_KERNEL(575); break;
		case 576: LAUNCH_KERNEL(576); break;
		case 577: LAUNCH_KERNEL(577); break;
		case 578: LAUNCH_KERNEL(578); break;
		case 579: LAUNCH_KERNEL(579); break;
		case 580: LAUNCH_KERNEL(580); break;
		case 581: LAUNCH_KERNEL(581); break;
		case 582: LAUNCH_KERNEL(582); break;
		case 583: LAUNCH_KERNEL(583); break;
		case 584: LAUNCH_KERNEL(584); break;
		case 585: LAUNCH_KERNEL(585); break;
		case 586: LAUNCH_KERNEL(586); break;
		case 587: LAUNCH_KERNEL(587); break;
		case 588: LAUNCH_KERNEL(588); break;
		case 589: LAUNCH_KERNEL(589); break;
		case 590: LAUNCH_KERNEL(590); break;
		case 591: LAUNCH_KERNEL(591); break;
		case 592: LAUNCH_KERNEL(592); break;
		case 593: LAUNCH_KERNEL(593); break;
		case 594: LAUNCH_KERNEL(594); break;
		case 595: LAUNCH_KERNEL(595); break;
		case 596: LAUNCH_KERNEL(596); break;
		case 597: LAUNCH_KERNEL(597); break;
		case 598: LAUNCH_KERNEL(598); break;
		case 599: LAUNCH_KERNEL(599); break;
		case 600: LAUNCH_KERNEL(600); break;
		case 601: LAUNCH_KERNEL(601); break;
		case 602: LAUNCH_KERNEL(602); break;
		case 603: LAUNCH_KERNEL(603); break;
		case 604: LAUNCH_KERNEL(604); break;
		case 605: LAUNCH_KERNEL(605); break;
		case 606: LAUNCH_KERNEL(606); break;
		case 607: LAUNCH_KERNEL(607); break;
		case 608: LAUNCH_KERNEL(608); break;
		case 609: LAUNCH_KERNEL(609); break;
		case 610: LAUNCH_KERNEL(610); break;
		case 611: LAUNCH_KERNEL(611); break;
		case 612: LAUNCH_KERNEL(612); break;
		case 613: LAUNCH_KERNEL(613); break;
		case 614: LAUNCH_KERNEL(614); break;
		case 615: LAUNCH_KERNEL(615); break;
		case 616: LAUNCH_KERNEL(616); break;
		case 617: LAUNCH_KERNEL(617); break;
		case 618: LAUNCH_KERNEL(618); break;
		case 619: LAUNCH_KERNEL(619); break;
		case 620: LAUNCH_KERNEL(620); break;
		case 621: LAUNCH_KERNEL(621); break;
		case 622: LAUNCH_KERNEL(622); break;
		case 623: LAUNCH_KERNEL(623); break;
		case 624: LAUNCH_KERNEL(624); break;
		case 625: LAUNCH_KERNEL(625); break;
		case 626: LAUNCH_KERNEL(626); break;
		case 627: LAUNCH_KERNEL(627); break;
		case 628: LAUNCH_KERNEL(628); break;
		case 629: LAUNCH_KERNEL(629); break;
		case 630: LAUNCH_KERNEL(630); break;
		case 631: LAUNCH_KERNEL(631); break;
		case 632: LAUNCH_KERNEL(632); break;
		case 633: LAUNCH_KERNEL(633); break;
		case 634: LAUNCH_KERNEL(634); break;
		case 635: LAUNCH_KERNEL(635); break;
		case 636: LAUNCH_KERNEL(636); break;
		case 637: LAUNCH_KERNEL(637); break;
		case 638: LAUNCH_KERNEL(638); break;
		case 639: LAUNCH_KERNEL(639); break;
		case 640: LAUNCH_KERNEL(640); break;
		case 641: LAUNCH_KERNEL(641); break;
		case 642: LAUNCH_KERNEL(642); break;
		case 643: LAUNCH_KERNEL(643); break;
		case 644: LAUNCH_KERNEL(644); break;
		case 645: LAUNCH_KERNEL(645); break;
		case 646: LAUNCH_KERNEL(646); break;
		case 647: LAUNCH_KERNEL(647); break;
		case 648: LAUNCH_KERNEL(648); break;
		case 649: LAUNCH_KERNEL(649); break;
		case 650: LAUNCH_KERNEL(650); break;
		case 651: LAUNCH_KERNEL(651); break;
		case 652: LAUNCH_KERNEL(652); break;
		case 653: LAUNCH_KERNEL(653); break;
		case 654: LAUNCH_KERNEL(654); break;
		case 655: LAUNCH_KERNEL(655); break;
		case 656: LAUNCH_KERNEL(656); break;
		case 657: LAUNCH_KERNEL(657); break;
		case 658: LAUNCH_KERNEL(658); break;
		case 659: LAUNCH_KERNEL(659); break;
		case 660: LAUNCH_KERNEL(660); break;
		case 661: LAUNCH_KERNEL(661); break;
		case 662: LAUNCH_KERNEL(662); break;
		case 663: LAUNCH_KERNEL(663); break;
		case 664: LAUNCH_KERNEL(664); break;
		case 665: LAUNCH_KERNEL(665); break;
		case 666: LAUNCH_KERNEL(666); break;
		case 667: LAUNCH_KERNEL(667); break;
		case 668: LAUNCH_KERNEL(668); break;
		case 669: LAUNCH_KERNEL(669); break;
		case 670: LAUNCH_KERNEL(670); break;
		case 671: LAUNCH_KERNEL(671); break;
		case 672: LAUNCH_KERNEL(672); break;
		case 673: LAUNCH_KERNEL(673); break;
		case 674: LAUNCH_KERNEL(674); break;
		case 675: LAUNCH_KERNEL(675); break;
		case 676: LAUNCH_KERNEL(676); break;
		case 677: LAUNCH_KERNEL(677); break;
		case 678: LAUNCH_KERNEL(678); break;
		case 679: LAUNCH_KERNEL(679); break;
		case 680: LAUNCH_KERNEL(680); break;
		case 681: LAUNCH_KERNEL(681); break;
		case 682: LAUNCH_KERNEL(682); break;
		case 683: LAUNCH_KERNEL(683); break;
		case 684: LAUNCH_KERNEL(684); break;
		case 685: LAUNCH_KERNEL(685); break;
		case 686: LAUNCH_KERNEL(686); break;
		case 687: LAUNCH_KERNEL(687); break;
		case 688: LAUNCH_KERNEL(688); break;
		case 689: LAUNCH_KERNEL(689); break;
		case 690: LAUNCH_KERNEL(690); break;
		case 691: LAUNCH_KERNEL(691); break;
		case 692: LAUNCH_KERNEL(692); break;
		case 693: LAUNCH_KERNEL(693); break;
		case 694: LAUNCH_KERNEL(694); break;
		case 695: LAUNCH_KERNEL(695); break;
		case 696: LAUNCH_KERNEL(696); break;
		case 697: LAUNCH_KERNEL(697); break;
		case 698: LAUNCH_KERNEL(698); break;
		case 699: LAUNCH_KERNEL(699); break;
		case 700: LAUNCH_KERNEL(700); break;
		case 701: LAUNCH_KERNEL(701); break;
		case 702: LAUNCH_KERNEL(702); break;
		case 703: LAUNCH_KERNEL(703); break;
		case 704: LAUNCH_KERNEL(704); break;
		case 705: LAUNCH_KERNEL(705); break;
		case 706: LAUNCH_KERNEL(706); break;
		case 707: LAUNCH_KERNEL(707); break;
		case 708: LAUNCH_KERNEL(708); break;
		case 709: LAUNCH_KERNEL(709); break;
		case 710: LAUNCH_KERNEL(710); break;
		case 711: LAUNCH_KERNEL(711); break;
		case 712: LAUNCH_KERNEL(712); break;
		case 713: LAUNCH_KERNEL(713); break;
		case 714: LAUNCH_KERNEL(714); break;
		case 715: LAUNCH_KERNEL(715); break;
		case 716: LAUNCH_KERNEL(716); break;
		case 717: LAUNCH_KERNEL(717); break;
		case 718: LAUNCH_KERNEL(718); break;
		case 719: LAUNCH_KERNEL(719); break;
		case 720: LAUNCH_KERNEL(720); break;
		case 721: LAUNCH_KERNEL(721); break;
		case 722: LAUNCH_KERNEL(722); break;
		case 723: LAUNCH_KERNEL(723); break;
		case 724: LAUNCH_KERNEL(724); break;
		case 725: LAUNCH_KERNEL(725); break;
		case 726: LAUNCH_KERNEL(726); break;
		case 727: LAUNCH_KERNEL(727); break;
		case 728: LAUNCH_KERNEL(728); break;
		case 729: LAUNCH_KERNEL(729); break;
		case 730: LAUNCH_KERNEL(730); break;
		case 731: LAUNCH_KERNEL(731); break;
		case 732: LAUNCH_KERNEL(732); break;
		case 733: LAUNCH_KERNEL(733); break;
		case 734: LAUNCH_KERNEL(734); break;
		case 735: LAUNCH_KERNEL(735); break;
		case 736: LAUNCH_KERNEL(736); break;
		case 737: LAUNCH_KERNEL(737); break;
		case 738: LAUNCH_KERNEL(738); break;
		case 739: LAUNCH_KERNEL(739); break;
		case 740: LAUNCH_KERNEL(740); break;
		case 741: LAUNCH_KERNEL(741); break;
		case 742: LAUNCH_KERNEL(742); break;
		case 743: LAUNCH_KERNEL(743); break;
		case 744: LAUNCH_KERNEL(744); break;
		case 745: LAUNCH_KERNEL(745); break;
		case 746: LAUNCH_KERNEL(746); break;
		case 747: LAUNCH_KERNEL(747); break;
		case 748: LAUNCH_KERNEL(748); break;
		case 749: LAUNCH_KERNEL(749); break;
		case 750: LAUNCH_KERNEL(750); break;
		case 751: LAUNCH_KERNEL(751); break;
		case 752: LAUNCH_KERNEL(752); break;
		case 753: LAUNCH_KERNEL(753); break;
		case 754: LAUNCH_KERNEL(754); break;
		case 755: LAUNCH_KERNEL(755); break;
		case 756: LAUNCH_KERNEL(756); break;
		case 757: LAUNCH_KERNEL(757); break;
		case 758: LAUNCH_KERNEL(758); break;
		case 759: LAUNCH_KERNEL(759); break;
		case 760: LAUNCH_KERNEL(760); break;
		case 761: LAUNCH_KERNEL(761); break;
		case 762: LAUNCH_KERNEL(762); break;
		case 763: LAUNCH_KERNEL(763); break;
		case 764: LAUNCH_KERNEL(764); break;
		case 765: LAUNCH_KERNEL(765); break;
		case 766: LAUNCH_KERNEL(766); break;
		case 767: LAUNCH_KERNEL(767); break;
		default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
