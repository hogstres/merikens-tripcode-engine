// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 3840
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3841
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3842
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3843
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3844
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3845
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3846
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3847
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3848
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3849
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3850
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3851
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3852
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3853
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3854
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3855
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3856
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3857
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3858
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3859
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3860
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3861
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3862
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3863
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3864
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3865
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3866
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3867
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3868
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3869
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3870
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3871
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3872
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3873
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3874
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3875
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3876
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3877
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3878
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3879
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3880
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3881
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3882
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3883
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3884
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3885
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3886
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3887
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3888
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3889
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3890
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3891
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3892
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3893
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3894
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3895
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3896
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3897
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3898
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3899
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3900
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3901
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3902
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3903
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3904
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3905
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3906
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3907
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3908
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3909
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3910
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3911
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3912
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3913
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3914
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3915
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3916
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3917
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3918
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3919
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3920
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3921
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3922
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3923
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3924
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3925
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3926
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3927
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3928
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3929
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3930
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3931
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3932
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3933
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3934
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3935
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3936
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3937
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3938
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3939
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3940
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3941
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3942
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3943
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3944
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3945
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3946
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3947
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3948
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3949
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3950
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3951
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3952
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3953
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3954
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3955
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3956
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3957
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3958
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3959
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3960
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3961
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3962
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3963
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3964
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3965
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3966
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3967
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3968
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3969
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3970
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3971
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3972
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3973
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3974
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3975
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3976
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3977
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3978
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3979
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3980
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3981
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3982
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3983
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3984
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3985
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3986
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3987
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3988
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3989
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3990
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3991
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3992
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3993
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3994
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3995
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3996
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3997
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3998
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3999
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4000
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4001
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4002
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4003
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4004
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4005
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4006
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4007
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4008
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4009
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4010
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4011
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4012
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4013
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4014
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4015
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4016
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4017
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4018
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4019
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4020
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4021
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4022
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4023
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4024
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4025
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4026
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4027
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4028
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4029
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4030
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4031
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4032
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4033
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4034
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4035
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4036
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4037
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4038
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4039
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4040
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4041
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4042
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4043
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4044
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4045
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4046
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4047
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4048
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4049
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4050
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4051
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4052
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4053
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4054
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4055
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4056
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4057
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4058
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4059
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4060
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4061
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4062
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4063
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4064
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4065
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4066
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4067
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4068
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4069
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4070
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4071
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4072
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4073
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4074
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4075
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4076
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4077
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4078
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4079
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4080
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4081
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4082
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4083
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4084
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4085
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4086
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4087
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4088
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4089
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4090
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4091
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4092
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4093
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4094
#include "../CUDA10_Registers_Kernel.h"
#define SALT 4095
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher15()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel15(
	uint32_t numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	uint32_t *cudaTripcodeChunkArray,
	uint32_t numTripcodeChunk,
	int32_t intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int32_t searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 3840: LAUNCH_KERNEL(3840); break;
	case 3841: LAUNCH_KERNEL(3841); break;
	case 3842: LAUNCH_KERNEL(3842); break;
	case 3843: LAUNCH_KERNEL(3843); break;
	case 3844: LAUNCH_KERNEL(3844); break;
	case 3845: LAUNCH_KERNEL(3845); break;
	case 3846: LAUNCH_KERNEL(3846); break;
	case 3847: LAUNCH_KERNEL(3847); break;
	case 3848: LAUNCH_KERNEL(3848); break;
	case 3849: LAUNCH_KERNEL(3849); break;
	case 3850: LAUNCH_KERNEL(3850); break;
	case 3851: LAUNCH_KERNEL(3851); break;
	case 3852: LAUNCH_KERNEL(3852); break;
	case 3853: LAUNCH_KERNEL(3853); break;
	case 3854: LAUNCH_KERNEL(3854); break;
	case 3855: LAUNCH_KERNEL(3855); break;
	case 3856: LAUNCH_KERNEL(3856); break;
	case 3857: LAUNCH_KERNEL(3857); break;
	case 3858: LAUNCH_KERNEL(3858); break;
	case 3859: LAUNCH_KERNEL(3859); break;
	case 3860: LAUNCH_KERNEL(3860); break;
	case 3861: LAUNCH_KERNEL(3861); break;
	case 3862: LAUNCH_KERNEL(3862); break;
	case 3863: LAUNCH_KERNEL(3863); break;
	case 3864: LAUNCH_KERNEL(3864); break;
	case 3865: LAUNCH_KERNEL(3865); break;
	case 3866: LAUNCH_KERNEL(3866); break;
	case 3867: LAUNCH_KERNEL(3867); break;
	case 3868: LAUNCH_KERNEL(3868); break;
	case 3869: LAUNCH_KERNEL(3869); break;
	case 3870: LAUNCH_KERNEL(3870); break;
	case 3871: LAUNCH_KERNEL(3871); break;
	case 3872: LAUNCH_KERNEL(3872); break;
	case 3873: LAUNCH_KERNEL(3873); break;
	case 3874: LAUNCH_KERNEL(3874); break;
	case 3875: LAUNCH_KERNEL(3875); break;
	case 3876: LAUNCH_KERNEL(3876); break;
	case 3877: LAUNCH_KERNEL(3877); break;
	case 3878: LAUNCH_KERNEL(3878); break;
	case 3879: LAUNCH_KERNEL(3879); break;
	case 3880: LAUNCH_KERNEL(3880); break;
	case 3881: LAUNCH_KERNEL(3881); break;
	case 3882: LAUNCH_KERNEL(3882); break;
	case 3883: LAUNCH_KERNEL(3883); break;
	case 3884: LAUNCH_KERNEL(3884); break;
	case 3885: LAUNCH_KERNEL(3885); break;
	case 3886: LAUNCH_KERNEL(3886); break;
	case 3887: LAUNCH_KERNEL(3887); break;
	case 3888: LAUNCH_KERNEL(3888); break;
	case 3889: LAUNCH_KERNEL(3889); break;
	case 3890: LAUNCH_KERNEL(3890); break;
	case 3891: LAUNCH_KERNEL(3891); break;
	case 3892: LAUNCH_KERNEL(3892); break;
	case 3893: LAUNCH_KERNEL(3893); break;
	case 3894: LAUNCH_KERNEL(3894); break;
	case 3895: LAUNCH_KERNEL(3895); break;
	case 3896: LAUNCH_KERNEL(3896); break;
	case 3897: LAUNCH_KERNEL(3897); break;
	case 3898: LAUNCH_KERNEL(3898); break;
	case 3899: LAUNCH_KERNEL(3899); break;
	case 3900: LAUNCH_KERNEL(3900); break;
	case 3901: LAUNCH_KERNEL(3901); break;
	case 3902: LAUNCH_KERNEL(3902); break;
	case 3903: LAUNCH_KERNEL(3903); break;
	case 3904: LAUNCH_KERNEL(3904); break;
	case 3905: LAUNCH_KERNEL(3905); break;
	case 3906: LAUNCH_KERNEL(3906); break;
	case 3907: LAUNCH_KERNEL(3907); break;
	case 3908: LAUNCH_KERNEL(3908); break;
	case 3909: LAUNCH_KERNEL(3909); break;
	case 3910: LAUNCH_KERNEL(3910); break;
	case 3911: LAUNCH_KERNEL(3911); break;
	case 3912: LAUNCH_KERNEL(3912); break;
	case 3913: LAUNCH_KERNEL(3913); break;
	case 3914: LAUNCH_KERNEL(3914); break;
	case 3915: LAUNCH_KERNEL(3915); break;
	case 3916: LAUNCH_KERNEL(3916); break;
	case 3917: LAUNCH_KERNEL(3917); break;
	case 3918: LAUNCH_KERNEL(3918); break;
	case 3919: LAUNCH_KERNEL(3919); break;
	case 3920: LAUNCH_KERNEL(3920); break;
	case 3921: LAUNCH_KERNEL(3921); break;
	case 3922: LAUNCH_KERNEL(3922); break;
	case 3923: LAUNCH_KERNEL(3923); break;
	case 3924: LAUNCH_KERNEL(3924); break;
	case 3925: LAUNCH_KERNEL(3925); break;
	case 3926: LAUNCH_KERNEL(3926); break;
	case 3927: LAUNCH_KERNEL(3927); break;
	case 3928: LAUNCH_KERNEL(3928); break;
	case 3929: LAUNCH_KERNEL(3929); break;
	case 3930: LAUNCH_KERNEL(3930); break;
	case 3931: LAUNCH_KERNEL(3931); break;
	case 3932: LAUNCH_KERNEL(3932); break;
	case 3933: LAUNCH_KERNEL(3933); break;
	case 3934: LAUNCH_KERNEL(3934); break;
	case 3935: LAUNCH_KERNEL(3935); break;
	case 3936: LAUNCH_KERNEL(3936); break;
	case 3937: LAUNCH_KERNEL(3937); break;
	case 3938: LAUNCH_KERNEL(3938); break;
	case 3939: LAUNCH_KERNEL(3939); break;
	case 3940: LAUNCH_KERNEL(3940); break;
	case 3941: LAUNCH_KERNEL(3941); break;
	case 3942: LAUNCH_KERNEL(3942); break;
	case 3943: LAUNCH_KERNEL(3943); break;
	case 3944: LAUNCH_KERNEL(3944); break;
	case 3945: LAUNCH_KERNEL(3945); break;
	case 3946: LAUNCH_KERNEL(3946); break;
	case 3947: LAUNCH_KERNEL(3947); break;
	case 3948: LAUNCH_KERNEL(3948); break;
	case 3949: LAUNCH_KERNEL(3949); break;
	case 3950: LAUNCH_KERNEL(3950); break;
	case 3951: LAUNCH_KERNEL(3951); break;
	case 3952: LAUNCH_KERNEL(3952); break;
	case 3953: LAUNCH_KERNEL(3953); break;
	case 3954: LAUNCH_KERNEL(3954); break;
	case 3955: LAUNCH_KERNEL(3955); break;
	case 3956: LAUNCH_KERNEL(3956); break;
	case 3957: LAUNCH_KERNEL(3957); break;
	case 3958: LAUNCH_KERNEL(3958); break;
	case 3959: LAUNCH_KERNEL(3959); break;
	case 3960: LAUNCH_KERNEL(3960); break;
	case 3961: LAUNCH_KERNEL(3961); break;
	case 3962: LAUNCH_KERNEL(3962); break;
	case 3963: LAUNCH_KERNEL(3963); break;
	case 3964: LAUNCH_KERNEL(3964); break;
	case 3965: LAUNCH_KERNEL(3965); break;
	case 3966: LAUNCH_KERNEL(3966); break;
	case 3967: LAUNCH_KERNEL(3967); break;
	case 3968: LAUNCH_KERNEL(3968); break;
	case 3969: LAUNCH_KERNEL(3969); break;
	case 3970: LAUNCH_KERNEL(3970); break;
	case 3971: LAUNCH_KERNEL(3971); break;
	case 3972: LAUNCH_KERNEL(3972); break;
	case 3973: LAUNCH_KERNEL(3973); break;
	case 3974: LAUNCH_KERNEL(3974); break;
	case 3975: LAUNCH_KERNEL(3975); break;
	case 3976: LAUNCH_KERNEL(3976); break;
	case 3977: LAUNCH_KERNEL(3977); break;
	case 3978: LAUNCH_KERNEL(3978); break;
	case 3979: LAUNCH_KERNEL(3979); break;
	case 3980: LAUNCH_KERNEL(3980); break;
	case 3981: LAUNCH_KERNEL(3981); break;
	case 3982: LAUNCH_KERNEL(3982); break;
	case 3983: LAUNCH_KERNEL(3983); break;
	case 3984: LAUNCH_KERNEL(3984); break;
	case 3985: LAUNCH_KERNEL(3985); break;
	case 3986: LAUNCH_KERNEL(3986); break;
	case 3987: LAUNCH_KERNEL(3987); break;
	case 3988: LAUNCH_KERNEL(3988); break;
	case 3989: LAUNCH_KERNEL(3989); break;
	case 3990: LAUNCH_KERNEL(3990); break;
	case 3991: LAUNCH_KERNEL(3991); break;
	case 3992: LAUNCH_KERNEL(3992); break;
	case 3993: LAUNCH_KERNEL(3993); break;
	case 3994: LAUNCH_KERNEL(3994); break;
	case 3995: LAUNCH_KERNEL(3995); break;
	case 3996: LAUNCH_KERNEL(3996); break;
	case 3997: LAUNCH_KERNEL(3997); break;
	case 3998: LAUNCH_KERNEL(3998); break;
	case 3999: LAUNCH_KERNEL(3999); break;
	case 4000: LAUNCH_KERNEL(4000); break;
	case 4001: LAUNCH_KERNEL(4001); break;
	case 4002: LAUNCH_KERNEL(4002); break;
	case 4003: LAUNCH_KERNEL(4003); break;
	case 4004: LAUNCH_KERNEL(4004); break;
	case 4005: LAUNCH_KERNEL(4005); break;
	case 4006: LAUNCH_KERNEL(4006); break;
	case 4007: LAUNCH_KERNEL(4007); break;
	case 4008: LAUNCH_KERNEL(4008); break;
	case 4009: LAUNCH_KERNEL(4009); break;
	case 4010: LAUNCH_KERNEL(4010); break;
	case 4011: LAUNCH_KERNEL(4011); break;
	case 4012: LAUNCH_KERNEL(4012); break;
	case 4013: LAUNCH_KERNEL(4013); break;
	case 4014: LAUNCH_KERNEL(4014); break;
	case 4015: LAUNCH_KERNEL(4015); break;
	case 4016: LAUNCH_KERNEL(4016); break;
	case 4017: LAUNCH_KERNEL(4017); break;
	case 4018: LAUNCH_KERNEL(4018); break;
	case 4019: LAUNCH_KERNEL(4019); break;
	case 4020: LAUNCH_KERNEL(4020); break;
	case 4021: LAUNCH_KERNEL(4021); break;
	case 4022: LAUNCH_KERNEL(4022); break;
	case 4023: LAUNCH_KERNEL(4023); break;
	case 4024: LAUNCH_KERNEL(4024); break;
	case 4025: LAUNCH_KERNEL(4025); break;
	case 4026: LAUNCH_KERNEL(4026); break;
	case 4027: LAUNCH_KERNEL(4027); break;
	case 4028: LAUNCH_KERNEL(4028); break;
	case 4029: LAUNCH_KERNEL(4029); break;
	case 4030: LAUNCH_KERNEL(4030); break;
	case 4031: LAUNCH_KERNEL(4031); break;
	case 4032: LAUNCH_KERNEL(4032); break;
	case 4033: LAUNCH_KERNEL(4033); break;
	case 4034: LAUNCH_KERNEL(4034); break;
	case 4035: LAUNCH_KERNEL(4035); break;
	case 4036: LAUNCH_KERNEL(4036); break;
	case 4037: LAUNCH_KERNEL(4037); break;
	case 4038: LAUNCH_KERNEL(4038); break;
	case 4039: LAUNCH_KERNEL(4039); break;
	case 4040: LAUNCH_KERNEL(4040); break;
	case 4041: LAUNCH_KERNEL(4041); break;
	case 4042: LAUNCH_KERNEL(4042); break;
	case 4043: LAUNCH_KERNEL(4043); break;
	case 4044: LAUNCH_KERNEL(4044); break;
	case 4045: LAUNCH_KERNEL(4045); break;
	case 4046: LAUNCH_KERNEL(4046); break;
	case 4047: LAUNCH_KERNEL(4047); break;
	case 4048: LAUNCH_KERNEL(4048); break;
	case 4049: LAUNCH_KERNEL(4049); break;
	case 4050: LAUNCH_KERNEL(4050); break;
	case 4051: LAUNCH_KERNEL(4051); break;
	case 4052: LAUNCH_KERNEL(4052); break;
	case 4053: LAUNCH_KERNEL(4053); break;
	case 4054: LAUNCH_KERNEL(4054); break;
	case 4055: LAUNCH_KERNEL(4055); break;
	case 4056: LAUNCH_KERNEL(4056); break;
	case 4057: LAUNCH_KERNEL(4057); break;
	case 4058: LAUNCH_KERNEL(4058); break;
	case 4059: LAUNCH_KERNEL(4059); break;
	case 4060: LAUNCH_KERNEL(4060); break;
	case 4061: LAUNCH_KERNEL(4061); break;
	case 4062: LAUNCH_KERNEL(4062); break;
	case 4063: LAUNCH_KERNEL(4063); break;
	case 4064: LAUNCH_KERNEL(4064); break;
	case 4065: LAUNCH_KERNEL(4065); break;
	case 4066: LAUNCH_KERNEL(4066); break;
	case 4067: LAUNCH_KERNEL(4067); break;
	case 4068: LAUNCH_KERNEL(4068); break;
	case 4069: LAUNCH_KERNEL(4069); break;
	case 4070: LAUNCH_KERNEL(4070); break;
	case 4071: LAUNCH_KERNEL(4071); break;
	case 4072: LAUNCH_KERNEL(4072); break;
	case 4073: LAUNCH_KERNEL(4073); break;
	case 4074: LAUNCH_KERNEL(4074); break;
	case 4075: LAUNCH_KERNEL(4075); break;
	case 4076: LAUNCH_KERNEL(4076); break;
	case 4077: LAUNCH_KERNEL(4077); break;
	case 4078: LAUNCH_KERNEL(4078); break;
	case 4079: LAUNCH_KERNEL(4079); break;
	case 4080: LAUNCH_KERNEL(4080); break;
	case 4081: LAUNCH_KERNEL(4081); break;
	case 4082: LAUNCH_KERNEL(4082); break;
	case 4083: LAUNCH_KERNEL(4083); break;
	case 4084: LAUNCH_KERNEL(4084); break;
	case 4085: LAUNCH_KERNEL(4085); break;
	case 4086: LAUNCH_KERNEL(4086); break;
	case 4087: LAUNCH_KERNEL(4087); break;
	case 4088: LAUNCH_KERNEL(4088); break;
	case 4089: LAUNCH_KERNEL(4089); break;
	case 4090: LAUNCH_KERNEL(4090); break;
	case 4091: LAUNCH_KERNEL(4091); break;
	case 4092: LAUNCH_KERNEL(4092); break;
	case 4093: LAUNCH_KERNEL(4093); break;
	case 4094: LAUNCH_KERNEL(4094); break;
	case 4095: LAUNCH_KERNEL(4095); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
