// Meriken's Tripcode Engine 2.0.1
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 1792
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1793
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1794
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1795
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1796
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1797
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1798
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1799
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1800
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1801
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1802
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1803
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1804
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1805
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1806
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1807
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1808
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1809
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1810
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1811
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1812
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1813
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1814
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1815
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1816
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1817
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1818
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1819
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1820
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1821
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1822
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1823
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1824
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1825
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1826
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1827
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1828
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1829
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1830
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1831
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1832
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1833
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1834
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1835
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1836
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1837
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1838
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1839
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1840
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1841
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1842
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1843
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1844
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1845
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1846
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1847
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1848
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1849
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1850
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1851
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1852
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1853
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1854
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1855
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1856
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1857
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1858
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1859
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1860
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1861
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1862
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1863
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1864
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1865
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1866
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1867
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1868
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1869
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1870
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1871
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1872
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1873
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1874
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1875
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1876
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1877
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1878
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1879
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1880
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1881
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1882
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1883
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1884
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1885
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1886
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1887
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1888
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1889
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1890
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1891
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1892
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1893
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1894
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1895
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1896
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1897
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1898
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1899
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1900
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1901
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1902
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1903
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1904
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1905
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1906
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1907
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1908
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1909
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1910
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1911
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1912
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1913
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1914
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1915
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1916
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1917
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1918
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1919
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1920
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1921
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1922
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1923
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1924
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1925
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1926
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1927
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1928
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1929
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1930
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1931
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1932
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1933
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1934
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1935
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1936
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1937
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1938
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1939
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1940
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1941
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1942
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1943
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1944
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1945
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1946
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1947
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1948
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1949
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1950
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1951
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1952
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1953
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1954
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1955
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1956
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1957
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1958
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1959
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1960
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1961
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1962
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1963
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1964
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1965
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1966
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1967
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1968
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1969
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1970
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1971
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1972
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1973
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1974
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1975
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1976
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1977
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1978
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1979
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1980
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1981
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1982
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1983
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1984
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1985
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1986
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1987
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1988
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1989
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1990
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1991
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1992
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1993
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1994
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1995
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1996
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1997
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1998
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1999
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2000
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2001
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2002
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2003
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2004
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2005
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2006
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2007
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2008
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2009
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2010
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2011
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2012
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2013
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2014
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2015
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2016
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2017
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2018
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2019
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2020
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2021
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2022
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2023
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2024
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2025
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2026
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2027
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2028
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2029
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2030
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2031
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2032
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2033
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2034
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2035
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2036
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2037
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2038
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2039
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2040
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2041
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2042
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2043
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2044
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2045
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2046
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2047
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher7()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel7(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 1792: LAUNCH_KERNEL(1792); break;
	case 1793: LAUNCH_KERNEL(1793); break;
	case 1794: LAUNCH_KERNEL(1794); break;
	case 1795: LAUNCH_KERNEL(1795); break;
	case 1796: LAUNCH_KERNEL(1796); break;
	case 1797: LAUNCH_KERNEL(1797); break;
	case 1798: LAUNCH_KERNEL(1798); break;
	case 1799: LAUNCH_KERNEL(1799); break;
	case 1800: LAUNCH_KERNEL(1800); break;
	case 1801: LAUNCH_KERNEL(1801); break;
	case 1802: LAUNCH_KERNEL(1802); break;
	case 1803: LAUNCH_KERNEL(1803); break;
	case 1804: LAUNCH_KERNEL(1804); break;
	case 1805: LAUNCH_KERNEL(1805); break;
	case 1806: LAUNCH_KERNEL(1806); break;
	case 1807: LAUNCH_KERNEL(1807); break;
	case 1808: LAUNCH_KERNEL(1808); break;
	case 1809: LAUNCH_KERNEL(1809); break;
	case 1810: LAUNCH_KERNEL(1810); break;
	case 1811: LAUNCH_KERNEL(1811); break;
	case 1812: LAUNCH_KERNEL(1812); break;
	case 1813: LAUNCH_KERNEL(1813); break;
	case 1814: LAUNCH_KERNEL(1814); break;
	case 1815: LAUNCH_KERNEL(1815); break;
	case 1816: LAUNCH_KERNEL(1816); break;
	case 1817: LAUNCH_KERNEL(1817); break;
	case 1818: LAUNCH_KERNEL(1818); break;
	case 1819: LAUNCH_KERNEL(1819); break;
	case 1820: LAUNCH_KERNEL(1820); break;
	case 1821: LAUNCH_KERNEL(1821); break;
	case 1822: LAUNCH_KERNEL(1822); break;
	case 1823: LAUNCH_KERNEL(1823); break;
	case 1824: LAUNCH_KERNEL(1824); break;
	case 1825: LAUNCH_KERNEL(1825); break;
	case 1826: LAUNCH_KERNEL(1826); break;
	case 1827: LAUNCH_KERNEL(1827); break;
	case 1828: LAUNCH_KERNEL(1828); break;
	case 1829: LAUNCH_KERNEL(1829); break;
	case 1830: LAUNCH_KERNEL(1830); break;
	case 1831: LAUNCH_KERNEL(1831); break;
	case 1832: LAUNCH_KERNEL(1832); break;
	case 1833: LAUNCH_KERNEL(1833); break;
	case 1834: LAUNCH_KERNEL(1834); break;
	case 1835: LAUNCH_KERNEL(1835); break;
	case 1836: LAUNCH_KERNEL(1836); break;
	case 1837: LAUNCH_KERNEL(1837); break;
	case 1838: LAUNCH_KERNEL(1838); break;
	case 1839: LAUNCH_KERNEL(1839); break;
	case 1840: LAUNCH_KERNEL(1840); break;
	case 1841: LAUNCH_KERNEL(1841); break;
	case 1842: LAUNCH_KERNEL(1842); break;
	case 1843: LAUNCH_KERNEL(1843); break;
	case 1844: LAUNCH_KERNEL(1844); break;
	case 1845: LAUNCH_KERNEL(1845); break;
	case 1846: LAUNCH_KERNEL(1846); break;
	case 1847: LAUNCH_KERNEL(1847); break;
	case 1848: LAUNCH_KERNEL(1848); break;
	case 1849: LAUNCH_KERNEL(1849); break;
	case 1850: LAUNCH_KERNEL(1850); break;
	case 1851: LAUNCH_KERNEL(1851); break;
	case 1852: LAUNCH_KERNEL(1852); break;
	case 1853: LAUNCH_KERNEL(1853); break;
	case 1854: LAUNCH_KERNEL(1854); break;
	case 1855: LAUNCH_KERNEL(1855); break;
	case 1856: LAUNCH_KERNEL(1856); break;
	case 1857: LAUNCH_KERNEL(1857); break;
	case 1858: LAUNCH_KERNEL(1858); break;
	case 1859: LAUNCH_KERNEL(1859); break;
	case 1860: LAUNCH_KERNEL(1860); break;
	case 1861: LAUNCH_KERNEL(1861); break;
	case 1862: LAUNCH_KERNEL(1862); break;
	case 1863: LAUNCH_KERNEL(1863); break;
	case 1864: LAUNCH_KERNEL(1864); break;
	case 1865: LAUNCH_KERNEL(1865); break;
	case 1866: LAUNCH_KERNEL(1866); break;
	case 1867: LAUNCH_KERNEL(1867); break;
	case 1868: LAUNCH_KERNEL(1868); break;
	case 1869: LAUNCH_KERNEL(1869); break;
	case 1870: LAUNCH_KERNEL(1870); break;
	case 1871: LAUNCH_KERNEL(1871); break;
	case 1872: LAUNCH_KERNEL(1872); break;
	case 1873: LAUNCH_KERNEL(1873); break;
	case 1874: LAUNCH_KERNEL(1874); break;
	case 1875: LAUNCH_KERNEL(1875); break;
	case 1876: LAUNCH_KERNEL(1876); break;
	case 1877: LAUNCH_KERNEL(1877); break;
	case 1878: LAUNCH_KERNEL(1878); break;
	case 1879: LAUNCH_KERNEL(1879); break;
	case 1880: LAUNCH_KERNEL(1880); break;
	case 1881: LAUNCH_KERNEL(1881); break;
	case 1882: LAUNCH_KERNEL(1882); break;
	case 1883: LAUNCH_KERNEL(1883); break;
	case 1884: LAUNCH_KERNEL(1884); break;
	case 1885: LAUNCH_KERNEL(1885); break;
	case 1886: LAUNCH_KERNEL(1886); break;
	case 1887: LAUNCH_KERNEL(1887); break;
	case 1888: LAUNCH_KERNEL(1888); break;
	case 1889: LAUNCH_KERNEL(1889); break;
	case 1890: LAUNCH_KERNEL(1890); break;
	case 1891: LAUNCH_KERNEL(1891); break;
	case 1892: LAUNCH_KERNEL(1892); break;
	case 1893: LAUNCH_KERNEL(1893); break;
	case 1894: LAUNCH_KERNEL(1894); break;
	case 1895: LAUNCH_KERNEL(1895); break;
	case 1896: LAUNCH_KERNEL(1896); break;
	case 1897: LAUNCH_KERNEL(1897); break;
	case 1898: LAUNCH_KERNEL(1898); break;
	case 1899: LAUNCH_KERNEL(1899); break;
	case 1900: LAUNCH_KERNEL(1900); break;
	case 1901: LAUNCH_KERNEL(1901); break;
	case 1902: LAUNCH_KERNEL(1902); break;
	case 1903: LAUNCH_KERNEL(1903); break;
	case 1904: LAUNCH_KERNEL(1904); break;
	case 1905: LAUNCH_KERNEL(1905); break;
	case 1906: LAUNCH_KERNEL(1906); break;
	case 1907: LAUNCH_KERNEL(1907); break;
	case 1908: LAUNCH_KERNEL(1908); break;
	case 1909: LAUNCH_KERNEL(1909); break;
	case 1910: LAUNCH_KERNEL(1910); break;
	case 1911: LAUNCH_KERNEL(1911); break;
	case 1912: LAUNCH_KERNEL(1912); break;
	case 1913: LAUNCH_KERNEL(1913); break;
	case 1914: LAUNCH_KERNEL(1914); break;
	case 1915: LAUNCH_KERNEL(1915); break;
	case 1916: LAUNCH_KERNEL(1916); break;
	case 1917: LAUNCH_KERNEL(1917); break;
	case 1918: LAUNCH_KERNEL(1918); break;
	case 1919: LAUNCH_KERNEL(1919); break;
	case 1920: LAUNCH_KERNEL(1920); break;
	case 1921: LAUNCH_KERNEL(1921); break;
	case 1922: LAUNCH_KERNEL(1922); break;
	case 1923: LAUNCH_KERNEL(1923); break;
	case 1924: LAUNCH_KERNEL(1924); break;
	case 1925: LAUNCH_KERNEL(1925); break;
	case 1926: LAUNCH_KERNEL(1926); break;
	case 1927: LAUNCH_KERNEL(1927); break;
	case 1928: LAUNCH_KERNEL(1928); break;
	case 1929: LAUNCH_KERNEL(1929); break;
	case 1930: LAUNCH_KERNEL(1930); break;
	case 1931: LAUNCH_KERNEL(1931); break;
	case 1932: LAUNCH_KERNEL(1932); break;
	case 1933: LAUNCH_KERNEL(1933); break;
	case 1934: LAUNCH_KERNEL(1934); break;
	case 1935: LAUNCH_KERNEL(1935); break;
	case 1936: LAUNCH_KERNEL(1936); break;
	case 1937: LAUNCH_KERNEL(1937); break;
	case 1938: LAUNCH_KERNEL(1938); break;
	case 1939: LAUNCH_KERNEL(1939); break;
	case 1940: LAUNCH_KERNEL(1940); break;
	case 1941: LAUNCH_KERNEL(1941); break;
	case 1942: LAUNCH_KERNEL(1942); break;
	case 1943: LAUNCH_KERNEL(1943); break;
	case 1944: LAUNCH_KERNEL(1944); break;
	case 1945: LAUNCH_KERNEL(1945); break;
	case 1946: LAUNCH_KERNEL(1946); break;
	case 1947: LAUNCH_KERNEL(1947); break;
	case 1948: LAUNCH_KERNEL(1948); break;
	case 1949: LAUNCH_KERNEL(1949); break;
	case 1950: LAUNCH_KERNEL(1950); break;
	case 1951: LAUNCH_KERNEL(1951); break;
	case 1952: LAUNCH_KERNEL(1952); break;
	case 1953: LAUNCH_KERNEL(1953); break;
	case 1954: LAUNCH_KERNEL(1954); break;
	case 1955: LAUNCH_KERNEL(1955); break;
	case 1956: LAUNCH_KERNEL(1956); break;
	case 1957: LAUNCH_KERNEL(1957); break;
	case 1958: LAUNCH_KERNEL(1958); break;
	case 1959: LAUNCH_KERNEL(1959); break;
	case 1960: LAUNCH_KERNEL(1960); break;
	case 1961: LAUNCH_KERNEL(1961); break;
	case 1962: LAUNCH_KERNEL(1962); break;
	case 1963: LAUNCH_KERNEL(1963); break;
	case 1964: LAUNCH_KERNEL(1964); break;
	case 1965: LAUNCH_KERNEL(1965); break;
	case 1966: LAUNCH_KERNEL(1966); break;
	case 1967: LAUNCH_KERNEL(1967); break;
	case 1968: LAUNCH_KERNEL(1968); break;
	case 1969: LAUNCH_KERNEL(1969); break;
	case 1970: LAUNCH_KERNEL(1970); break;
	case 1971: LAUNCH_KERNEL(1971); break;
	case 1972: LAUNCH_KERNEL(1972); break;
	case 1973: LAUNCH_KERNEL(1973); break;
	case 1974: LAUNCH_KERNEL(1974); break;
	case 1975: LAUNCH_KERNEL(1975); break;
	case 1976: LAUNCH_KERNEL(1976); break;
	case 1977: LAUNCH_KERNEL(1977); break;
	case 1978: LAUNCH_KERNEL(1978); break;
	case 1979: LAUNCH_KERNEL(1979); break;
	case 1980: LAUNCH_KERNEL(1980); break;
	case 1981: LAUNCH_KERNEL(1981); break;
	case 1982: LAUNCH_KERNEL(1982); break;
	case 1983: LAUNCH_KERNEL(1983); break;
	case 1984: LAUNCH_KERNEL(1984); break;
	case 1985: LAUNCH_KERNEL(1985); break;
	case 1986: LAUNCH_KERNEL(1986); break;
	case 1987: LAUNCH_KERNEL(1987); break;
	case 1988: LAUNCH_KERNEL(1988); break;
	case 1989: LAUNCH_KERNEL(1989); break;
	case 1990: LAUNCH_KERNEL(1990); break;
	case 1991: LAUNCH_KERNEL(1991); break;
	case 1992: LAUNCH_KERNEL(1992); break;
	case 1993: LAUNCH_KERNEL(1993); break;
	case 1994: LAUNCH_KERNEL(1994); break;
	case 1995: LAUNCH_KERNEL(1995); break;
	case 1996: LAUNCH_KERNEL(1996); break;
	case 1997: LAUNCH_KERNEL(1997); break;
	case 1998: LAUNCH_KERNEL(1998); break;
	case 1999: LAUNCH_KERNEL(1999); break;
	case 2000: LAUNCH_KERNEL(2000); break;
	case 2001: LAUNCH_KERNEL(2001); break;
	case 2002: LAUNCH_KERNEL(2002); break;
	case 2003: LAUNCH_KERNEL(2003); break;
	case 2004: LAUNCH_KERNEL(2004); break;
	case 2005: LAUNCH_KERNEL(2005); break;
	case 2006: LAUNCH_KERNEL(2006); break;
	case 2007: LAUNCH_KERNEL(2007); break;
	case 2008: LAUNCH_KERNEL(2008); break;
	case 2009: LAUNCH_KERNEL(2009); break;
	case 2010: LAUNCH_KERNEL(2010); break;
	case 2011: LAUNCH_KERNEL(2011); break;
	case 2012: LAUNCH_KERNEL(2012); break;
	case 2013: LAUNCH_KERNEL(2013); break;
	case 2014: LAUNCH_KERNEL(2014); break;
	case 2015: LAUNCH_KERNEL(2015); break;
	case 2016: LAUNCH_KERNEL(2016); break;
	case 2017: LAUNCH_KERNEL(2017); break;
	case 2018: LAUNCH_KERNEL(2018); break;
	case 2019: LAUNCH_KERNEL(2019); break;
	case 2020: LAUNCH_KERNEL(2020); break;
	case 2021: LAUNCH_KERNEL(2021); break;
	case 2022: LAUNCH_KERNEL(2022); break;
	case 2023: LAUNCH_KERNEL(2023); break;
	case 2024: LAUNCH_KERNEL(2024); break;
	case 2025: LAUNCH_KERNEL(2025); break;
	case 2026: LAUNCH_KERNEL(2026); break;
	case 2027: LAUNCH_KERNEL(2027); break;
	case 2028: LAUNCH_KERNEL(2028); break;
	case 2029: LAUNCH_KERNEL(2029); break;
	case 2030: LAUNCH_KERNEL(2030); break;
	case 2031: LAUNCH_KERNEL(2031); break;
	case 2032: LAUNCH_KERNEL(2032); break;
	case 2033: LAUNCH_KERNEL(2033); break;
	case 2034: LAUNCH_KERNEL(2034); break;
	case 2035: LAUNCH_KERNEL(2035); break;
	case 2036: LAUNCH_KERNEL(2036); break;
	case 2037: LAUNCH_KERNEL(2037); break;
	case 2038: LAUNCH_KERNEL(2038); break;
	case 2039: LAUNCH_KERNEL(2039); break;
	case 2040: LAUNCH_KERNEL(2040); break;
	case 2041: LAUNCH_KERNEL(2041); break;
	case 2042: LAUNCH_KERNEL(2042); break;
	case 2043: LAUNCH_KERNEL(2043); break;
	case 2044: LAUNCH_KERNEL(2044); break;
	case 2045: LAUNCH_KERNEL(2045); break;
	case 2046: LAUNCH_KERNEL(2046); break;
	case 2047: LAUNCH_KERNEL(2047); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
