// Meriken's Tripcode Engine
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 1280
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1281
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1282
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1283
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1284
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1285
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1286
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1287
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1288
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1289
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1290
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1291
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1292
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1293
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1294
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1295
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1296
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1297
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1298
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1299
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1300
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1301
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1302
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1303
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1304
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1305
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1306
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1307
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1308
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1309
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1310
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1311
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1312
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1313
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1314
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1315
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1316
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1317
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1318
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1319
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1320
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1321
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1322
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1323
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1324
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1325
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1326
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1327
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1328
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1329
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1330
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1331
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1332
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1333
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1334
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1335
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1336
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1337
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1338
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1339
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1340
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1341
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1342
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1343
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1344
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1345
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1346
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1347
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1348
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1349
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1350
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1351
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1352
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1353
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1354
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1355
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1356
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1357
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1358
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1359
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1360
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1361
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1362
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1363
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1364
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1365
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1366
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1367
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1368
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1369
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1370
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1371
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1372
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1373
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1374
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1375
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1376
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1377
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1378
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1379
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1380
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1381
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1382
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1383
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1384
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1385
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1386
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1387
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1388
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1389
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1390
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1391
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1392
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1393
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1394
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1395
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1396
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1397
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1398
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1399
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1400
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1401
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1402
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1403
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1404
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1405
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1406
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1407
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1408
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1409
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1410
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1411
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1412
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1413
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1414
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1415
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1416
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1417
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1418
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1419
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1420
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1421
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1422
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1423
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1424
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1425
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1426
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1427
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1428
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1429
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1430
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1431
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1432
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1433
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1434
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1435
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1436
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1437
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1438
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1439
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1440
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1441
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1442
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1443
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1444
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1445
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1446
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1447
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1448
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1449
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1450
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1451
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1452
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1453
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1454
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1455
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1456
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1457
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1458
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1459
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1460
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1461
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1462
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1463
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1464
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1465
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1466
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1467
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1468
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1469
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1470
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1471
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1472
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1473
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1474
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1475
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1476
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1477
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1478
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1479
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1480
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1481
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1482
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1483
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1484
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1485
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1486
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1487
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1488
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1489
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1490
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1491
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1492
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1493
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1494
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1495
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1496
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1497
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1498
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1499
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1500
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1501
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1502
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1503
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1504
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1505
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1506
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1507
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1508
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1509
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1510
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1511
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1512
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1513
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1514
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1515
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1516
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1517
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1518
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1519
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1520
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1521
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1522
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1523
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1524
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1525
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1526
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1527
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1528
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1529
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1530
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1531
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1532
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1533
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1534
#include "../CUDA10_Registers_Kernel.h"
#define SALT 1535
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher5()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel5(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 1280: LAUNCH_KERNEL(1280); break;
	case 1281: LAUNCH_KERNEL(1281); break;
	case 1282: LAUNCH_KERNEL(1282); break;
	case 1283: LAUNCH_KERNEL(1283); break;
	case 1284: LAUNCH_KERNEL(1284); break;
	case 1285: LAUNCH_KERNEL(1285); break;
	case 1286: LAUNCH_KERNEL(1286); break;
	case 1287: LAUNCH_KERNEL(1287); break;
	case 1288: LAUNCH_KERNEL(1288); break;
	case 1289: LAUNCH_KERNEL(1289); break;
	case 1290: LAUNCH_KERNEL(1290); break;
	case 1291: LAUNCH_KERNEL(1291); break;
	case 1292: LAUNCH_KERNEL(1292); break;
	case 1293: LAUNCH_KERNEL(1293); break;
	case 1294: LAUNCH_KERNEL(1294); break;
	case 1295: LAUNCH_KERNEL(1295); break;
	case 1296: LAUNCH_KERNEL(1296); break;
	case 1297: LAUNCH_KERNEL(1297); break;
	case 1298: LAUNCH_KERNEL(1298); break;
	case 1299: LAUNCH_KERNEL(1299); break;
	case 1300: LAUNCH_KERNEL(1300); break;
	case 1301: LAUNCH_KERNEL(1301); break;
	case 1302: LAUNCH_KERNEL(1302); break;
	case 1303: LAUNCH_KERNEL(1303); break;
	case 1304: LAUNCH_KERNEL(1304); break;
	case 1305: LAUNCH_KERNEL(1305); break;
	case 1306: LAUNCH_KERNEL(1306); break;
	case 1307: LAUNCH_KERNEL(1307); break;
	case 1308: LAUNCH_KERNEL(1308); break;
	case 1309: LAUNCH_KERNEL(1309); break;
	case 1310: LAUNCH_KERNEL(1310); break;
	case 1311: LAUNCH_KERNEL(1311); break;
	case 1312: LAUNCH_KERNEL(1312); break;
	case 1313: LAUNCH_KERNEL(1313); break;
	case 1314: LAUNCH_KERNEL(1314); break;
	case 1315: LAUNCH_KERNEL(1315); break;
	case 1316: LAUNCH_KERNEL(1316); break;
	case 1317: LAUNCH_KERNEL(1317); break;
	case 1318: LAUNCH_KERNEL(1318); break;
	case 1319: LAUNCH_KERNEL(1319); break;
	case 1320: LAUNCH_KERNEL(1320); break;
	case 1321: LAUNCH_KERNEL(1321); break;
	case 1322: LAUNCH_KERNEL(1322); break;
	case 1323: LAUNCH_KERNEL(1323); break;
	case 1324: LAUNCH_KERNEL(1324); break;
	case 1325: LAUNCH_KERNEL(1325); break;
	case 1326: LAUNCH_KERNEL(1326); break;
	case 1327: LAUNCH_KERNEL(1327); break;
	case 1328: LAUNCH_KERNEL(1328); break;
	case 1329: LAUNCH_KERNEL(1329); break;
	case 1330: LAUNCH_KERNEL(1330); break;
	case 1331: LAUNCH_KERNEL(1331); break;
	case 1332: LAUNCH_KERNEL(1332); break;
	case 1333: LAUNCH_KERNEL(1333); break;
	case 1334: LAUNCH_KERNEL(1334); break;
	case 1335: LAUNCH_KERNEL(1335); break;
	case 1336: LAUNCH_KERNEL(1336); break;
	case 1337: LAUNCH_KERNEL(1337); break;
	case 1338: LAUNCH_KERNEL(1338); break;
	case 1339: LAUNCH_KERNEL(1339); break;
	case 1340: LAUNCH_KERNEL(1340); break;
	case 1341: LAUNCH_KERNEL(1341); break;
	case 1342: LAUNCH_KERNEL(1342); break;
	case 1343: LAUNCH_KERNEL(1343); break;
	case 1344: LAUNCH_KERNEL(1344); break;
	case 1345: LAUNCH_KERNEL(1345); break;
	case 1346: LAUNCH_KERNEL(1346); break;
	case 1347: LAUNCH_KERNEL(1347); break;
	case 1348: LAUNCH_KERNEL(1348); break;
	case 1349: LAUNCH_KERNEL(1349); break;
	case 1350: LAUNCH_KERNEL(1350); break;
	case 1351: LAUNCH_KERNEL(1351); break;
	case 1352: LAUNCH_KERNEL(1352); break;
	case 1353: LAUNCH_KERNEL(1353); break;
	case 1354: LAUNCH_KERNEL(1354); break;
	case 1355: LAUNCH_KERNEL(1355); break;
	case 1356: LAUNCH_KERNEL(1356); break;
	case 1357: LAUNCH_KERNEL(1357); break;
	case 1358: LAUNCH_KERNEL(1358); break;
	case 1359: LAUNCH_KERNEL(1359); break;
	case 1360: LAUNCH_KERNEL(1360); break;
	case 1361: LAUNCH_KERNEL(1361); break;
	case 1362: LAUNCH_KERNEL(1362); break;
	case 1363: LAUNCH_KERNEL(1363); break;
	case 1364: LAUNCH_KERNEL(1364); break;
	case 1365: LAUNCH_KERNEL(1365); break;
	case 1366: LAUNCH_KERNEL(1366); break;
	case 1367: LAUNCH_KERNEL(1367); break;
	case 1368: LAUNCH_KERNEL(1368); break;
	case 1369: LAUNCH_KERNEL(1369); break;
	case 1370: LAUNCH_KERNEL(1370); break;
	case 1371: LAUNCH_KERNEL(1371); break;
	case 1372: LAUNCH_KERNEL(1372); break;
	case 1373: LAUNCH_KERNEL(1373); break;
	case 1374: LAUNCH_KERNEL(1374); break;
	case 1375: LAUNCH_KERNEL(1375); break;
	case 1376: LAUNCH_KERNEL(1376); break;
	case 1377: LAUNCH_KERNEL(1377); break;
	case 1378: LAUNCH_KERNEL(1378); break;
	case 1379: LAUNCH_KERNEL(1379); break;
	case 1380: LAUNCH_KERNEL(1380); break;
	case 1381: LAUNCH_KERNEL(1381); break;
	case 1382: LAUNCH_KERNEL(1382); break;
	case 1383: LAUNCH_KERNEL(1383); break;
	case 1384: LAUNCH_KERNEL(1384); break;
	case 1385: LAUNCH_KERNEL(1385); break;
	case 1386: LAUNCH_KERNEL(1386); break;
	case 1387: LAUNCH_KERNEL(1387); break;
	case 1388: LAUNCH_KERNEL(1388); break;
	case 1389: LAUNCH_KERNEL(1389); break;
	case 1390: LAUNCH_KERNEL(1390); break;
	case 1391: LAUNCH_KERNEL(1391); break;
	case 1392: LAUNCH_KERNEL(1392); break;
	case 1393: LAUNCH_KERNEL(1393); break;
	case 1394: LAUNCH_KERNEL(1394); break;
	case 1395: LAUNCH_KERNEL(1395); break;
	case 1396: LAUNCH_KERNEL(1396); break;
	case 1397: LAUNCH_KERNEL(1397); break;
	case 1398: LAUNCH_KERNEL(1398); break;
	case 1399: LAUNCH_KERNEL(1399); break;
	case 1400: LAUNCH_KERNEL(1400); break;
	case 1401: LAUNCH_KERNEL(1401); break;
	case 1402: LAUNCH_KERNEL(1402); break;
	case 1403: LAUNCH_KERNEL(1403); break;
	case 1404: LAUNCH_KERNEL(1404); break;
	case 1405: LAUNCH_KERNEL(1405); break;
	case 1406: LAUNCH_KERNEL(1406); break;
	case 1407: LAUNCH_KERNEL(1407); break;
	case 1408: LAUNCH_KERNEL(1408); break;
	case 1409: LAUNCH_KERNEL(1409); break;
	case 1410: LAUNCH_KERNEL(1410); break;
	case 1411: LAUNCH_KERNEL(1411); break;
	case 1412: LAUNCH_KERNEL(1412); break;
	case 1413: LAUNCH_KERNEL(1413); break;
	case 1414: LAUNCH_KERNEL(1414); break;
	case 1415: LAUNCH_KERNEL(1415); break;
	case 1416: LAUNCH_KERNEL(1416); break;
	case 1417: LAUNCH_KERNEL(1417); break;
	case 1418: LAUNCH_KERNEL(1418); break;
	case 1419: LAUNCH_KERNEL(1419); break;
	case 1420: LAUNCH_KERNEL(1420); break;
	case 1421: LAUNCH_KERNEL(1421); break;
	case 1422: LAUNCH_KERNEL(1422); break;
	case 1423: LAUNCH_KERNEL(1423); break;
	case 1424: LAUNCH_KERNEL(1424); break;
	case 1425: LAUNCH_KERNEL(1425); break;
	case 1426: LAUNCH_KERNEL(1426); break;
	case 1427: LAUNCH_KERNEL(1427); break;
	case 1428: LAUNCH_KERNEL(1428); break;
	case 1429: LAUNCH_KERNEL(1429); break;
	case 1430: LAUNCH_KERNEL(1430); break;
	case 1431: LAUNCH_KERNEL(1431); break;
	case 1432: LAUNCH_KERNEL(1432); break;
	case 1433: LAUNCH_KERNEL(1433); break;
	case 1434: LAUNCH_KERNEL(1434); break;
	case 1435: LAUNCH_KERNEL(1435); break;
	case 1436: LAUNCH_KERNEL(1436); break;
	case 1437: LAUNCH_KERNEL(1437); break;
	case 1438: LAUNCH_KERNEL(1438); break;
	case 1439: LAUNCH_KERNEL(1439); break;
	case 1440: LAUNCH_KERNEL(1440); break;
	case 1441: LAUNCH_KERNEL(1441); break;
	case 1442: LAUNCH_KERNEL(1442); break;
	case 1443: LAUNCH_KERNEL(1443); break;
	case 1444: LAUNCH_KERNEL(1444); break;
	case 1445: LAUNCH_KERNEL(1445); break;
	case 1446: LAUNCH_KERNEL(1446); break;
	case 1447: LAUNCH_KERNEL(1447); break;
	case 1448: LAUNCH_KERNEL(1448); break;
	case 1449: LAUNCH_KERNEL(1449); break;
	case 1450: LAUNCH_KERNEL(1450); break;
	case 1451: LAUNCH_KERNEL(1451); break;
	case 1452: LAUNCH_KERNEL(1452); break;
	case 1453: LAUNCH_KERNEL(1453); break;
	case 1454: LAUNCH_KERNEL(1454); break;
	case 1455: LAUNCH_KERNEL(1455); break;
	case 1456: LAUNCH_KERNEL(1456); break;
	case 1457: LAUNCH_KERNEL(1457); break;
	case 1458: LAUNCH_KERNEL(1458); break;
	case 1459: LAUNCH_KERNEL(1459); break;
	case 1460: LAUNCH_KERNEL(1460); break;
	case 1461: LAUNCH_KERNEL(1461); break;
	case 1462: LAUNCH_KERNEL(1462); break;
	case 1463: LAUNCH_KERNEL(1463); break;
	case 1464: LAUNCH_KERNEL(1464); break;
	case 1465: LAUNCH_KERNEL(1465); break;
	case 1466: LAUNCH_KERNEL(1466); break;
	case 1467: LAUNCH_KERNEL(1467); break;
	case 1468: LAUNCH_KERNEL(1468); break;
	case 1469: LAUNCH_KERNEL(1469); break;
	case 1470: LAUNCH_KERNEL(1470); break;
	case 1471: LAUNCH_KERNEL(1471); break;
	case 1472: LAUNCH_KERNEL(1472); break;
	case 1473: LAUNCH_KERNEL(1473); break;
	case 1474: LAUNCH_KERNEL(1474); break;
	case 1475: LAUNCH_KERNEL(1475); break;
	case 1476: LAUNCH_KERNEL(1476); break;
	case 1477: LAUNCH_KERNEL(1477); break;
	case 1478: LAUNCH_KERNEL(1478); break;
	case 1479: LAUNCH_KERNEL(1479); break;
	case 1480: LAUNCH_KERNEL(1480); break;
	case 1481: LAUNCH_KERNEL(1481); break;
	case 1482: LAUNCH_KERNEL(1482); break;
	case 1483: LAUNCH_KERNEL(1483); break;
	case 1484: LAUNCH_KERNEL(1484); break;
	case 1485: LAUNCH_KERNEL(1485); break;
	case 1486: LAUNCH_KERNEL(1486); break;
	case 1487: LAUNCH_KERNEL(1487); break;
	case 1488: LAUNCH_KERNEL(1488); break;
	case 1489: LAUNCH_KERNEL(1489); break;
	case 1490: LAUNCH_KERNEL(1490); break;
	case 1491: LAUNCH_KERNEL(1491); break;
	case 1492: LAUNCH_KERNEL(1492); break;
	case 1493: LAUNCH_KERNEL(1493); break;
	case 1494: LAUNCH_KERNEL(1494); break;
	case 1495: LAUNCH_KERNEL(1495); break;
	case 1496: LAUNCH_KERNEL(1496); break;
	case 1497: LAUNCH_KERNEL(1497); break;
	case 1498: LAUNCH_KERNEL(1498); break;
	case 1499: LAUNCH_KERNEL(1499); break;
	case 1500: LAUNCH_KERNEL(1500); break;
	case 1501: LAUNCH_KERNEL(1501); break;
	case 1502: LAUNCH_KERNEL(1502); break;
	case 1503: LAUNCH_KERNEL(1503); break;
	case 1504: LAUNCH_KERNEL(1504); break;
	case 1505: LAUNCH_KERNEL(1505); break;
	case 1506: LAUNCH_KERNEL(1506); break;
	case 1507: LAUNCH_KERNEL(1507); break;
	case 1508: LAUNCH_KERNEL(1508); break;
	case 1509: LAUNCH_KERNEL(1509); break;
	case 1510: LAUNCH_KERNEL(1510); break;
	case 1511: LAUNCH_KERNEL(1511); break;
	case 1512: LAUNCH_KERNEL(1512); break;
	case 1513: LAUNCH_KERNEL(1513); break;
	case 1514: LAUNCH_KERNEL(1514); break;
	case 1515: LAUNCH_KERNEL(1515); break;
	case 1516: LAUNCH_KERNEL(1516); break;
	case 1517: LAUNCH_KERNEL(1517); break;
	case 1518: LAUNCH_KERNEL(1518); break;
	case 1519: LAUNCH_KERNEL(1519); break;
	case 1520: LAUNCH_KERNEL(1520); break;
	case 1521: LAUNCH_KERNEL(1521); break;
	case 1522: LAUNCH_KERNEL(1522); break;
	case 1523: LAUNCH_KERNEL(1523); break;
	case 1524: LAUNCH_KERNEL(1524); break;
	case 1525: LAUNCH_KERNEL(1525); break;
	case 1526: LAUNCH_KERNEL(1526); break;
	case 1527: LAUNCH_KERNEL(1527); break;
	case 1528: LAUNCH_KERNEL(1528); break;
	case 1529: LAUNCH_KERNEL(1529); break;
	case 1530: LAUNCH_KERNEL(1530); break;
	case 1531: LAUNCH_KERNEL(1531); break;
	case 1532: LAUNCH_KERNEL(1532); break;
	case 1533: LAUNCH_KERNEL(1533); break;
	case 1534: LAUNCH_KERNEL(1534); break;
	case 1535: LAUNCH_KERNEL(1535); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
