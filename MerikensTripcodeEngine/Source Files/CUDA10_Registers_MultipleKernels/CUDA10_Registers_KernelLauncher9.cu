// Meriken's Tripcode Engine 2.0.1
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 2304
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2305
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2306
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2307
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2308
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2309
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2310
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2311
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2312
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2313
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2314
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2315
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2316
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2317
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2318
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2319
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2320
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2321
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2322
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2323
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2324
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2325
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2326
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2327
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2328
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2329
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2330
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2331
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2332
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2333
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2334
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2335
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2336
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2337
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2338
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2339
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2340
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2341
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2342
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2343
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2344
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2345
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2346
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2347
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2348
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2349
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2350
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2351
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2352
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2353
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2354
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2355
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2356
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2357
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2358
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2359
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2360
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2361
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2362
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2363
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2364
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2365
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2366
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2367
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2368
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2369
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2370
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2371
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2372
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2373
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2374
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2375
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2376
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2377
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2378
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2379
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2380
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2381
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2382
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2383
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2384
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2385
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2386
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2387
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2388
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2389
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2390
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2391
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2392
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2393
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2394
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2395
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2396
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2397
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2398
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2399
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2400
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2401
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2402
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2403
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2404
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2405
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2406
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2407
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2408
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2409
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2410
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2411
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2412
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2413
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2414
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2415
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2416
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2417
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2418
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2419
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2420
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2421
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2422
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2423
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2424
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2425
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2426
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2427
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2428
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2429
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2430
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2431
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2432
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2433
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2434
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2435
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2436
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2437
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2438
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2439
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2440
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2441
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2442
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2443
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2444
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2445
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2446
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2447
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2448
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2449
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2450
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2451
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2452
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2453
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2454
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2455
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2456
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2457
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2458
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2459
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2460
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2461
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2462
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2463
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2464
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2465
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2466
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2467
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2468
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2469
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2470
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2471
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2472
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2473
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2474
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2475
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2476
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2477
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2478
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2479
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2480
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2481
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2482
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2483
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2484
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2485
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2486
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2487
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2488
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2489
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2490
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2491
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2492
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2493
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2494
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2495
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2496
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2497
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2498
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2499
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2500
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2501
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2502
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2503
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2504
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2505
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2506
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2507
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2508
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2509
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2510
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2511
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2512
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2513
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2514
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2515
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2516
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2517
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2518
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2519
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2520
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2521
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2522
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2523
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2524
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2525
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2526
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2527
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2528
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2529
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2530
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2531
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2532
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2533
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2534
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2535
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2536
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2537
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2538
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2539
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2540
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2541
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2542
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2543
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2544
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2545
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2546
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2547
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2548
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2549
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2550
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2551
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2552
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2553
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2554
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2555
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2556
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2557
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2558
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2559
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher9()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel9(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 2304: LAUNCH_KERNEL(2304); break;
	case 2305: LAUNCH_KERNEL(2305); break;
	case 2306: LAUNCH_KERNEL(2306); break;
	case 2307: LAUNCH_KERNEL(2307); break;
	case 2308: LAUNCH_KERNEL(2308); break;
	case 2309: LAUNCH_KERNEL(2309); break;
	case 2310: LAUNCH_KERNEL(2310); break;
	case 2311: LAUNCH_KERNEL(2311); break;
	case 2312: LAUNCH_KERNEL(2312); break;
	case 2313: LAUNCH_KERNEL(2313); break;
	case 2314: LAUNCH_KERNEL(2314); break;
	case 2315: LAUNCH_KERNEL(2315); break;
	case 2316: LAUNCH_KERNEL(2316); break;
	case 2317: LAUNCH_KERNEL(2317); break;
	case 2318: LAUNCH_KERNEL(2318); break;
	case 2319: LAUNCH_KERNEL(2319); break;
	case 2320: LAUNCH_KERNEL(2320); break;
	case 2321: LAUNCH_KERNEL(2321); break;
	case 2322: LAUNCH_KERNEL(2322); break;
	case 2323: LAUNCH_KERNEL(2323); break;
	case 2324: LAUNCH_KERNEL(2324); break;
	case 2325: LAUNCH_KERNEL(2325); break;
	case 2326: LAUNCH_KERNEL(2326); break;
	case 2327: LAUNCH_KERNEL(2327); break;
	case 2328: LAUNCH_KERNEL(2328); break;
	case 2329: LAUNCH_KERNEL(2329); break;
	case 2330: LAUNCH_KERNEL(2330); break;
	case 2331: LAUNCH_KERNEL(2331); break;
	case 2332: LAUNCH_KERNEL(2332); break;
	case 2333: LAUNCH_KERNEL(2333); break;
	case 2334: LAUNCH_KERNEL(2334); break;
	case 2335: LAUNCH_KERNEL(2335); break;
	case 2336: LAUNCH_KERNEL(2336); break;
	case 2337: LAUNCH_KERNEL(2337); break;
	case 2338: LAUNCH_KERNEL(2338); break;
	case 2339: LAUNCH_KERNEL(2339); break;
	case 2340: LAUNCH_KERNEL(2340); break;
	case 2341: LAUNCH_KERNEL(2341); break;
	case 2342: LAUNCH_KERNEL(2342); break;
	case 2343: LAUNCH_KERNEL(2343); break;
	case 2344: LAUNCH_KERNEL(2344); break;
	case 2345: LAUNCH_KERNEL(2345); break;
	case 2346: LAUNCH_KERNEL(2346); break;
	case 2347: LAUNCH_KERNEL(2347); break;
	case 2348: LAUNCH_KERNEL(2348); break;
	case 2349: LAUNCH_KERNEL(2349); break;
	case 2350: LAUNCH_KERNEL(2350); break;
	case 2351: LAUNCH_KERNEL(2351); break;
	case 2352: LAUNCH_KERNEL(2352); break;
	case 2353: LAUNCH_KERNEL(2353); break;
	case 2354: LAUNCH_KERNEL(2354); break;
	case 2355: LAUNCH_KERNEL(2355); break;
	case 2356: LAUNCH_KERNEL(2356); break;
	case 2357: LAUNCH_KERNEL(2357); break;
	case 2358: LAUNCH_KERNEL(2358); break;
	case 2359: LAUNCH_KERNEL(2359); break;
	case 2360: LAUNCH_KERNEL(2360); break;
	case 2361: LAUNCH_KERNEL(2361); break;
	case 2362: LAUNCH_KERNEL(2362); break;
	case 2363: LAUNCH_KERNEL(2363); break;
	case 2364: LAUNCH_KERNEL(2364); break;
	case 2365: LAUNCH_KERNEL(2365); break;
	case 2366: LAUNCH_KERNEL(2366); break;
	case 2367: LAUNCH_KERNEL(2367); break;
	case 2368: LAUNCH_KERNEL(2368); break;
	case 2369: LAUNCH_KERNEL(2369); break;
	case 2370: LAUNCH_KERNEL(2370); break;
	case 2371: LAUNCH_KERNEL(2371); break;
	case 2372: LAUNCH_KERNEL(2372); break;
	case 2373: LAUNCH_KERNEL(2373); break;
	case 2374: LAUNCH_KERNEL(2374); break;
	case 2375: LAUNCH_KERNEL(2375); break;
	case 2376: LAUNCH_KERNEL(2376); break;
	case 2377: LAUNCH_KERNEL(2377); break;
	case 2378: LAUNCH_KERNEL(2378); break;
	case 2379: LAUNCH_KERNEL(2379); break;
	case 2380: LAUNCH_KERNEL(2380); break;
	case 2381: LAUNCH_KERNEL(2381); break;
	case 2382: LAUNCH_KERNEL(2382); break;
	case 2383: LAUNCH_KERNEL(2383); break;
	case 2384: LAUNCH_KERNEL(2384); break;
	case 2385: LAUNCH_KERNEL(2385); break;
	case 2386: LAUNCH_KERNEL(2386); break;
	case 2387: LAUNCH_KERNEL(2387); break;
	case 2388: LAUNCH_KERNEL(2388); break;
	case 2389: LAUNCH_KERNEL(2389); break;
	case 2390: LAUNCH_KERNEL(2390); break;
	case 2391: LAUNCH_KERNEL(2391); break;
	case 2392: LAUNCH_KERNEL(2392); break;
	case 2393: LAUNCH_KERNEL(2393); break;
	case 2394: LAUNCH_KERNEL(2394); break;
	case 2395: LAUNCH_KERNEL(2395); break;
	case 2396: LAUNCH_KERNEL(2396); break;
	case 2397: LAUNCH_KERNEL(2397); break;
	case 2398: LAUNCH_KERNEL(2398); break;
	case 2399: LAUNCH_KERNEL(2399); break;
	case 2400: LAUNCH_KERNEL(2400); break;
	case 2401: LAUNCH_KERNEL(2401); break;
	case 2402: LAUNCH_KERNEL(2402); break;
	case 2403: LAUNCH_KERNEL(2403); break;
	case 2404: LAUNCH_KERNEL(2404); break;
	case 2405: LAUNCH_KERNEL(2405); break;
	case 2406: LAUNCH_KERNEL(2406); break;
	case 2407: LAUNCH_KERNEL(2407); break;
	case 2408: LAUNCH_KERNEL(2408); break;
	case 2409: LAUNCH_KERNEL(2409); break;
	case 2410: LAUNCH_KERNEL(2410); break;
	case 2411: LAUNCH_KERNEL(2411); break;
	case 2412: LAUNCH_KERNEL(2412); break;
	case 2413: LAUNCH_KERNEL(2413); break;
	case 2414: LAUNCH_KERNEL(2414); break;
	case 2415: LAUNCH_KERNEL(2415); break;
	case 2416: LAUNCH_KERNEL(2416); break;
	case 2417: LAUNCH_KERNEL(2417); break;
	case 2418: LAUNCH_KERNEL(2418); break;
	case 2419: LAUNCH_KERNEL(2419); break;
	case 2420: LAUNCH_KERNEL(2420); break;
	case 2421: LAUNCH_KERNEL(2421); break;
	case 2422: LAUNCH_KERNEL(2422); break;
	case 2423: LAUNCH_KERNEL(2423); break;
	case 2424: LAUNCH_KERNEL(2424); break;
	case 2425: LAUNCH_KERNEL(2425); break;
	case 2426: LAUNCH_KERNEL(2426); break;
	case 2427: LAUNCH_KERNEL(2427); break;
	case 2428: LAUNCH_KERNEL(2428); break;
	case 2429: LAUNCH_KERNEL(2429); break;
	case 2430: LAUNCH_KERNEL(2430); break;
	case 2431: LAUNCH_KERNEL(2431); break;
	case 2432: LAUNCH_KERNEL(2432); break;
	case 2433: LAUNCH_KERNEL(2433); break;
	case 2434: LAUNCH_KERNEL(2434); break;
	case 2435: LAUNCH_KERNEL(2435); break;
	case 2436: LAUNCH_KERNEL(2436); break;
	case 2437: LAUNCH_KERNEL(2437); break;
	case 2438: LAUNCH_KERNEL(2438); break;
	case 2439: LAUNCH_KERNEL(2439); break;
	case 2440: LAUNCH_KERNEL(2440); break;
	case 2441: LAUNCH_KERNEL(2441); break;
	case 2442: LAUNCH_KERNEL(2442); break;
	case 2443: LAUNCH_KERNEL(2443); break;
	case 2444: LAUNCH_KERNEL(2444); break;
	case 2445: LAUNCH_KERNEL(2445); break;
	case 2446: LAUNCH_KERNEL(2446); break;
	case 2447: LAUNCH_KERNEL(2447); break;
	case 2448: LAUNCH_KERNEL(2448); break;
	case 2449: LAUNCH_KERNEL(2449); break;
	case 2450: LAUNCH_KERNEL(2450); break;
	case 2451: LAUNCH_KERNEL(2451); break;
	case 2452: LAUNCH_KERNEL(2452); break;
	case 2453: LAUNCH_KERNEL(2453); break;
	case 2454: LAUNCH_KERNEL(2454); break;
	case 2455: LAUNCH_KERNEL(2455); break;
	case 2456: LAUNCH_KERNEL(2456); break;
	case 2457: LAUNCH_KERNEL(2457); break;
	case 2458: LAUNCH_KERNEL(2458); break;
	case 2459: LAUNCH_KERNEL(2459); break;
	case 2460: LAUNCH_KERNEL(2460); break;
	case 2461: LAUNCH_KERNEL(2461); break;
	case 2462: LAUNCH_KERNEL(2462); break;
	case 2463: LAUNCH_KERNEL(2463); break;
	case 2464: LAUNCH_KERNEL(2464); break;
	case 2465: LAUNCH_KERNEL(2465); break;
	case 2466: LAUNCH_KERNEL(2466); break;
	case 2467: LAUNCH_KERNEL(2467); break;
	case 2468: LAUNCH_KERNEL(2468); break;
	case 2469: LAUNCH_KERNEL(2469); break;
	case 2470: LAUNCH_KERNEL(2470); break;
	case 2471: LAUNCH_KERNEL(2471); break;
	case 2472: LAUNCH_KERNEL(2472); break;
	case 2473: LAUNCH_KERNEL(2473); break;
	case 2474: LAUNCH_KERNEL(2474); break;
	case 2475: LAUNCH_KERNEL(2475); break;
	case 2476: LAUNCH_KERNEL(2476); break;
	case 2477: LAUNCH_KERNEL(2477); break;
	case 2478: LAUNCH_KERNEL(2478); break;
	case 2479: LAUNCH_KERNEL(2479); break;
	case 2480: LAUNCH_KERNEL(2480); break;
	case 2481: LAUNCH_KERNEL(2481); break;
	case 2482: LAUNCH_KERNEL(2482); break;
	case 2483: LAUNCH_KERNEL(2483); break;
	case 2484: LAUNCH_KERNEL(2484); break;
	case 2485: LAUNCH_KERNEL(2485); break;
	case 2486: LAUNCH_KERNEL(2486); break;
	case 2487: LAUNCH_KERNEL(2487); break;
	case 2488: LAUNCH_KERNEL(2488); break;
	case 2489: LAUNCH_KERNEL(2489); break;
	case 2490: LAUNCH_KERNEL(2490); break;
	case 2491: LAUNCH_KERNEL(2491); break;
	case 2492: LAUNCH_KERNEL(2492); break;
	case 2493: LAUNCH_KERNEL(2493); break;
	case 2494: LAUNCH_KERNEL(2494); break;
	case 2495: LAUNCH_KERNEL(2495); break;
	case 2496: LAUNCH_KERNEL(2496); break;
	case 2497: LAUNCH_KERNEL(2497); break;
	case 2498: LAUNCH_KERNEL(2498); break;
	case 2499: LAUNCH_KERNEL(2499); break;
	case 2500: LAUNCH_KERNEL(2500); break;
	case 2501: LAUNCH_KERNEL(2501); break;
	case 2502: LAUNCH_KERNEL(2502); break;
	case 2503: LAUNCH_KERNEL(2503); break;
	case 2504: LAUNCH_KERNEL(2504); break;
	case 2505: LAUNCH_KERNEL(2505); break;
	case 2506: LAUNCH_KERNEL(2506); break;
	case 2507: LAUNCH_KERNEL(2507); break;
	case 2508: LAUNCH_KERNEL(2508); break;
	case 2509: LAUNCH_KERNEL(2509); break;
	case 2510: LAUNCH_KERNEL(2510); break;
	case 2511: LAUNCH_KERNEL(2511); break;
	case 2512: LAUNCH_KERNEL(2512); break;
	case 2513: LAUNCH_KERNEL(2513); break;
	case 2514: LAUNCH_KERNEL(2514); break;
	case 2515: LAUNCH_KERNEL(2515); break;
	case 2516: LAUNCH_KERNEL(2516); break;
	case 2517: LAUNCH_KERNEL(2517); break;
	case 2518: LAUNCH_KERNEL(2518); break;
	case 2519: LAUNCH_KERNEL(2519); break;
	case 2520: LAUNCH_KERNEL(2520); break;
	case 2521: LAUNCH_KERNEL(2521); break;
	case 2522: LAUNCH_KERNEL(2522); break;
	case 2523: LAUNCH_KERNEL(2523); break;
	case 2524: LAUNCH_KERNEL(2524); break;
	case 2525: LAUNCH_KERNEL(2525); break;
	case 2526: LAUNCH_KERNEL(2526); break;
	case 2527: LAUNCH_KERNEL(2527); break;
	case 2528: LAUNCH_KERNEL(2528); break;
	case 2529: LAUNCH_KERNEL(2529); break;
	case 2530: LAUNCH_KERNEL(2530); break;
	case 2531: LAUNCH_KERNEL(2531); break;
	case 2532: LAUNCH_KERNEL(2532); break;
	case 2533: LAUNCH_KERNEL(2533); break;
	case 2534: LAUNCH_KERNEL(2534); break;
	case 2535: LAUNCH_KERNEL(2535); break;
	case 2536: LAUNCH_KERNEL(2536); break;
	case 2537: LAUNCH_KERNEL(2537); break;
	case 2538: LAUNCH_KERNEL(2538); break;
	case 2539: LAUNCH_KERNEL(2539); break;
	case 2540: LAUNCH_KERNEL(2540); break;
	case 2541: LAUNCH_KERNEL(2541); break;
	case 2542: LAUNCH_KERNEL(2542); break;
	case 2543: LAUNCH_KERNEL(2543); break;
	case 2544: LAUNCH_KERNEL(2544); break;
	case 2545: LAUNCH_KERNEL(2545); break;
	case 2546: LAUNCH_KERNEL(2546); break;
	case 2547: LAUNCH_KERNEL(2547); break;
	case 2548: LAUNCH_KERNEL(2548); break;
	case 2549: LAUNCH_KERNEL(2549); break;
	case 2550: LAUNCH_KERNEL(2550); break;
	case 2551: LAUNCH_KERNEL(2551); break;
	case 2552: LAUNCH_KERNEL(2552); break;
	case 2553: LAUNCH_KERNEL(2553); break;
	case 2554: LAUNCH_KERNEL(2554); break;
	case 2555: LAUNCH_KERNEL(2555); break;
	case 2556: LAUNCH_KERNEL(2556); break;
	case 2557: LAUNCH_KERNEL(2557); break;
	case 2558: LAUNCH_KERNEL(2558); break;
	case 2559: LAUNCH_KERNEL(2559); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
