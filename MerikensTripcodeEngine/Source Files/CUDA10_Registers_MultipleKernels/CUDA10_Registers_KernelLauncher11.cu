// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 2816
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2817
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2818
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2819
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2820
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2821
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2822
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2823
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2824
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2825
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2826
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2827
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2828
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2829
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2830
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2831
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2832
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2833
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2834
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2835
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2836
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2837
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2838
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2839
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2840
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2841
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2842
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2843
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2844
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2845
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2846
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2847
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2848
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2849
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2850
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2851
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2852
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2853
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2854
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2855
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2856
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2857
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2858
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2859
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2860
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2861
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2862
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2863
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2864
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2865
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2866
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2867
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2868
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2869
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2870
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2871
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2872
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2873
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2874
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2875
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2876
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2877
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2878
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2879
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2880
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2881
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2882
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2883
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2884
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2885
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2886
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2887
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2888
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2889
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2890
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2891
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2892
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2893
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2894
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2895
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2896
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2897
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2898
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2899
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2900
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2901
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2902
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2903
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2904
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2905
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2906
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2907
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2908
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2909
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2910
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2911
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2912
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2913
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2914
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2915
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2916
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2917
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2918
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2919
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2920
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2921
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2922
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2923
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2924
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2925
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2926
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2927
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2928
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2929
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2930
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2931
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2932
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2933
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2934
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2935
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2936
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2937
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2938
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2939
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2940
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2941
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2942
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2943
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2944
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2945
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2946
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2947
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2948
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2949
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2950
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2951
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2952
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2953
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2954
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2955
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2956
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2957
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2958
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2959
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2960
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2961
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2962
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2963
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2964
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2965
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2966
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2967
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2968
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2969
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2970
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2971
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2972
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2973
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2974
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2975
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2976
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2977
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2978
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2979
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2980
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2981
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2982
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2983
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2984
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2985
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2986
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2987
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2988
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2989
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2990
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2991
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2992
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2993
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2994
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2995
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2996
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2997
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2998
#include "../CUDA10_Registers_Kernel.h"
#define SALT 2999
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3000
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3001
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3002
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3003
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3004
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3005
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3006
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3007
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3008
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3009
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3010
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3011
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3012
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3013
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3014
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3015
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3016
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3017
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3018
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3019
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3020
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3021
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3022
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3023
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3024
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3025
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3026
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3027
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3028
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3029
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3030
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3031
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3032
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3033
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3034
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3035
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3036
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3037
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3038
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3039
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3040
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3041
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3042
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3043
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3044
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3045
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3046
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3047
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3048
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3049
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3050
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3051
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3052
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3053
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3054
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3055
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3056
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3057
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3058
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3059
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3060
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3061
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3062
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3063
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3064
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3065
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3066
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3067
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3068
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3069
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3070
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3071
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher11()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel11(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 2816: LAUNCH_KERNEL(2816); break;
	case 2817: LAUNCH_KERNEL(2817); break;
	case 2818: LAUNCH_KERNEL(2818); break;
	case 2819: LAUNCH_KERNEL(2819); break;
	case 2820: LAUNCH_KERNEL(2820); break;
	case 2821: LAUNCH_KERNEL(2821); break;
	case 2822: LAUNCH_KERNEL(2822); break;
	case 2823: LAUNCH_KERNEL(2823); break;
	case 2824: LAUNCH_KERNEL(2824); break;
	case 2825: LAUNCH_KERNEL(2825); break;
	case 2826: LAUNCH_KERNEL(2826); break;
	case 2827: LAUNCH_KERNEL(2827); break;
	case 2828: LAUNCH_KERNEL(2828); break;
	case 2829: LAUNCH_KERNEL(2829); break;
	case 2830: LAUNCH_KERNEL(2830); break;
	case 2831: LAUNCH_KERNEL(2831); break;
	case 2832: LAUNCH_KERNEL(2832); break;
	case 2833: LAUNCH_KERNEL(2833); break;
	case 2834: LAUNCH_KERNEL(2834); break;
	case 2835: LAUNCH_KERNEL(2835); break;
	case 2836: LAUNCH_KERNEL(2836); break;
	case 2837: LAUNCH_KERNEL(2837); break;
	case 2838: LAUNCH_KERNEL(2838); break;
	case 2839: LAUNCH_KERNEL(2839); break;
	case 2840: LAUNCH_KERNEL(2840); break;
	case 2841: LAUNCH_KERNEL(2841); break;
	case 2842: LAUNCH_KERNEL(2842); break;
	case 2843: LAUNCH_KERNEL(2843); break;
	case 2844: LAUNCH_KERNEL(2844); break;
	case 2845: LAUNCH_KERNEL(2845); break;
	case 2846: LAUNCH_KERNEL(2846); break;
	case 2847: LAUNCH_KERNEL(2847); break;
	case 2848: LAUNCH_KERNEL(2848); break;
	case 2849: LAUNCH_KERNEL(2849); break;
	case 2850: LAUNCH_KERNEL(2850); break;
	case 2851: LAUNCH_KERNEL(2851); break;
	case 2852: LAUNCH_KERNEL(2852); break;
	case 2853: LAUNCH_KERNEL(2853); break;
	case 2854: LAUNCH_KERNEL(2854); break;
	case 2855: LAUNCH_KERNEL(2855); break;
	case 2856: LAUNCH_KERNEL(2856); break;
	case 2857: LAUNCH_KERNEL(2857); break;
	case 2858: LAUNCH_KERNEL(2858); break;
	case 2859: LAUNCH_KERNEL(2859); break;
	case 2860: LAUNCH_KERNEL(2860); break;
	case 2861: LAUNCH_KERNEL(2861); break;
	case 2862: LAUNCH_KERNEL(2862); break;
	case 2863: LAUNCH_KERNEL(2863); break;
	case 2864: LAUNCH_KERNEL(2864); break;
	case 2865: LAUNCH_KERNEL(2865); break;
	case 2866: LAUNCH_KERNEL(2866); break;
	case 2867: LAUNCH_KERNEL(2867); break;
	case 2868: LAUNCH_KERNEL(2868); break;
	case 2869: LAUNCH_KERNEL(2869); break;
	case 2870: LAUNCH_KERNEL(2870); break;
	case 2871: LAUNCH_KERNEL(2871); break;
	case 2872: LAUNCH_KERNEL(2872); break;
	case 2873: LAUNCH_KERNEL(2873); break;
	case 2874: LAUNCH_KERNEL(2874); break;
	case 2875: LAUNCH_KERNEL(2875); break;
	case 2876: LAUNCH_KERNEL(2876); break;
	case 2877: LAUNCH_KERNEL(2877); break;
	case 2878: LAUNCH_KERNEL(2878); break;
	case 2879: LAUNCH_KERNEL(2879); break;
	case 2880: LAUNCH_KERNEL(2880); break;
	case 2881: LAUNCH_KERNEL(2881); break;
	case 2882: LAUNCH_KERNEL(2882); break;
	case 2883: LAUNCH_KERNEL(2883); break;
	case 2884: LAUNCH_KERNEL(2884); break;
	case 2885: LAUNCH_KERNEL(2885); break;
	case 2886: LAUNCH_KERNEL(2886); break;
	case 2887: LAUNCH_KERNEL(2887); break;
	case 2888: LAUNCH_KERNEL(2888); break;
	case 2889: LAUNCH_KERNEL(2889); break;
	case 2890: LAUNCH_KERNEL(2890); break;
	case 2891: LAUNCH_KERNEL(2891); break;
	case 2892: LAUNCH_KERNEL(2892); break;
	case 2893: LAUNCH_KERNEL(2893); break;
	case 2894: LAUNCH_KERNEL(2894); break;
	case 2895: LAUNCH_KERNEL(2895); break;
	case 2896: LAUNCH_KERNEL(2896); break;
	case 2897: LAUNCH_KERNEL(2897); break;
	case 2898: LAUNCH_KERNEL(2898); break;
	case 2899: LAUNCH_KERNEL(2899); break;
	case 2900: LAUNCH_KERNEL(2900); break;
	case 2901: LAUNCH_KERNEL(2901); break;
	case 2902: LAUNCH_KERNEL(2902); break;
	case 2903: LAUNCH_KERNEL(2903); break;
	case 2904: LAUNCH_KERNEL(2904); break;
	case 2905: LAUNCH_KERNEL(2905); break;
	case 2906: LAUNCH_KERNEL(2906); break;
	case 2907: LAUNCH_KERNEL(2907); break;
	case 2908: LAUNCH_KERNEL(2908); break;
	case 2909: LAUNCH_KERNEL(2909); break;
	case 2910: LAUNCH_KERNEL(2910); break;
	case 2911: LAUNCH_KERNEL(2911); break;
	case 2912: LAUNCH_KERNEL(2912); break;
	case 2913: LAUNCH_KERNEL(2913); break;
	case 2914: LAUNCH_KERNEL(2914); break;
	case 2915: LAUNCH_KERNEL(2915); break;
	case 2916: LAUNCH_KERNEL(2916); break;
	case 2917: LAUNCH_KERNEL(2917); break;
	case 2918: LAUNCH_KERNEL(2918); break;
	case 2919: LAUNCH_KERNEL(2919); break;
	case 2920: LAUNCH_KERNEL(2920); break;
	case 2921: LAUNCH_KERNEL(2921); break;
	case 2922: LAUNCH_KERNEL(2922); break;
	case 2923: LAUNCH_KERNEL(2923); break;
	case 2924: LAUNCH_KERNEL(2924); break;
	case 2925: LAUNCH_KERNEL(2925); break;
	case 2926: LAUNCH_KERNEL(2926); break;
	case 2927: LAUNCH_KERNEL(2927); break;
	case 2928: LAUNCH_KERNEL(2928); break;
	case 2929: LAUNCH_KERNEL(2929); break;
	case 2930: LAUNCH_KERNEL(2930); break;
	case 2931: LAUNCH_KERNEL(2931); break;
	case 2932: LAUNCH_KERNEL(2932); break;
	case 2933: LAUNCH_KERNEL(2933); break;
	case 2934: LAUNCH_KERNEL(2934); break;
	case 2935: LAUNCH_KERNEL(2935); break;
	case 2936: LAUNCH_KERNEL(2936); break;
	case 2937: LAUNCH_KERNEL(2937); break;
	case 2938: LAUNCH_KERNEL(2938); break;
	case 2939: LAUNCH_KERNEL(2939); break;
	case 2940: LAUNCH_KERNEL(2940); break;
	case 2941: LAUNCH_KERNEL(2941); break;
	case 2942: LAUNCH_KERNEL(2942); break;
	case 2943: LAUNCH_KERNEL(2943); break;
	case 2944: LAUNCH_KERNEL(2944); break;
	case 2945: LAUNCH_KERNEL(2945); break;
	case 2946: LAUNCH_KERNEL(2946); break;
	case 2947: LAUNCH_KERNEL(2947); break;
	case 2948: LAUNCH_KERNEL(2948); break;
	case 2949: LAUNCH_KERNEL(2949); break;
	case 2950: LAUNCH_KERNEL(2950); break;
	case 2951: LAUNCH_KERNEL(2951); break;
	case 2952: LAUNCH_KERNEL(2952); break;
	case 2953: LAUNCH_KERNEL(2953); break;
	case 2954: LAUNCH_KERNEL(2954); break;
	case 2955: LAUNCH_KERNEL(2955); break;
	case 2956: LAUNCH_KERNEL(2956); break;
	case 2957: LAUNCH_KERNEL(2957); break;
	case 2958: LAUNCH_KERNEL(2958); break;
	case 2959: LAUNCH_KERNEL(2959); break;
	case 2960: LAUNCH_KERNEL(2960); break;
	case 2961: LAUNCH_KERNEL(2961); break;
	case 2962: LAUNCH_KERNEL(2962); break;
	case 2963: LAUNCH_KERNEL(2963); break;
	case 2964: LAUNCH_KERNEL(2964); break;
	case 2965: LAUNCH_KERNEL(2965); break;
	case 2966: LAUNCH_KERNEL(2966); break;
	case 2967: LAUNCH_KERNEL(2967); break;
	case 2968: LAUNCH_KERNEL(2968); break;
	case 2969: LAUNCH_KERNEL(2969); break;
	case 2970: LAUNCH_KERNEL(2970); break;
	case 2971: LAUNCH_KERNEL(2971); break;
	case 2972: LAUNCH_KERNEL(2972); break;
	case 2973: LAUNCH_KERNEL(2973); break;
	case 2974: LAUNCH_KERNEL(2974); break;
	case 2975: LAUNCH_KERNEL(2975); break;
	case 2976: LAUNCH_KERNEL(2976); break;
	case 2977: LAUNCH_KERNEL(2977); break;
	case 2978: LAUNCH_KERNEL(2978); break;
	case 2979: LAUNCH_KERNEL(2979); break;
	case 2980: LAUNCH_KERNEL(2980); break;
	case 2981: LAUNCH_KERNEL(2981); break;
	case 2982: LAUNCH_KERNEL(2982); break;
	case 2983: LAUNCH_KERNEL(2983); break;
	case 2984: LAUNCH_KERNEL(2984); break;
	case 2985: LAUNCH_KERNEL(2985); break;
	case 2986: LAUNCH_KERNEL(2986); break;
	case 2987: LAUNCH_KERNEL(2987); break;
	case 2988: LAUNCH_KERNEL(2988); break;
	case 2989: LAUNCH_KERNEL(2989); break;
	case 2990: LAUNCH_KERNEL(2990); break;
	case 2991: LAUNCH_KERNEL(2991); break;
	case 2992: LAUNCH_KERNEL(2992); break;
	case 2993: LAUNCH_KERNEL(2993); break;
	case 2994: LAUNCH_KERNEL(2994); break;
	case 2995: LAUNCH_KERNEL(2995); break;
	case 2996: LAUNCH_KERNEL(2996); break;
	case 2997: LAUNCH_KERNEL(2997); break;
	case 2998: LAUNCH_KERNEL(2998); break;
	case 2999: LAUNCH_KERNEL(2999); break;
	case 3000: LAUNCH_KERNEL(3000); break;
	case 3001: LAUNCH_KERNEL(3001); break;
	case 3002: LAUNCH_KERNEL(3002); break;
	case 3003: LAUNCH_KERNEL(3003); break;
	case 3004: LAUNCH_KERNEL(3004); break;
	case 3005: LAUNCH_KERNEL(3005); break;
	case 3006: LAUNCH_KERNEL(3006); break;
	case 3007: LAUNCH_KERNEL(3007); break;
	case 3008: LAUNCH_KERNEL(3008); break;
	case 3009: LAUNCH_KERNEL(3009); break;
	case 3010: LAUNCH_KERNEL(3010); break;
	case 3011: LAUNCH_KERNEL(3011); break;
	case 3012: LAUNCH_KERNEL(3012); break;
	case 3013: LAUNCH_KERNEL(3013); break;
	case 3014: LAUNCH_KERNEL(3014); break;
	case 3015: LAUNCH_KERNEL(3015); break;
	case 3016: LAUNCH_KERNEL(3016); break;
	case 3017: LAUNCH_KERNEL(3017); break;
	case 3018: LAUNCH_KERNEL(3018); break;
	case 3019: LAUNCH_KERNEL(3019); break;
	case 3020: LAUNCH_KERNEL(3020); break;
	case 3021: LAUNCH_KERNEL(3021); break;
	case 3022: LAUNCH_KERNEL(3022); break;
	case 3023: LAUNCH_KERNEL(3023); break;
	case 3024: LAUNCH_KERNEL(3024); break;
	case 3025: LAUNCH_KERNEL(3025); break;
	case 3026: LAUNCH_KERNEL(3026); break;
	case 3027: LAUNCH_KERNEL(3027); break;
	case 3028: LAUNCH_KERNEL(3028); break;
	case 3029: LAUNCH_KERNEL(3029); break;
	case 3030: LAUNCH_KERNEL(3030); break;
	case 3031: LAUNCH_KERNEL(3031); break;
	case 3032: LAUNCH_KERNEL(3032); break;
	case 3033: LAUNCH_KERNEL(3033); break;
	case 3034: LAUNCH_KERNEL(3034); break;
	case 3035: LAUNCH_KERNEL(3035); break;
	case 3036: LAUNCH_KERNEL(3036); break;
	case 3037: LAUNCH_KERNEL(3037); break;
	case 3038: LAUNCH_KERNEL(3038); break;
	case 3039: LAUNCH_KERNEL(3039); break;
	case 3040: LAUNCH_KERNEL(3040); break;
	case 3041: LAUNCH_KERNEL(3041); break;
	case 3042: LAUNCH_KERNEL(3042); break;
	case 3043: LAUNCH_KERNEL(3043); break;
	case 3044: LAUNCH_KERNEL(3044); break;
	case 3045: LAUNCH_KERNEL(3045); break;
	case 3046: LAUNCH_KERNEL(3046); break;
	case 3047: LAUNCH_KERNEL(3047); break;
	case 3048: LAUNCH_KERNEL(3048); break;
	case 3049: LAUNCH_KERNEL(3049); break;
	case 3050: LAUNCH_KERNEL(3050); break;
	case 3051: LAUNCH_KERNEL(3051); break;
	case 3052: LAUNCH_KERNEL(3052); break;
	case 3053: LAUNCH_KERNEL(3053); break;
	case 3054: LAUNCH_KERNEL(3054); break;
	case 3055: LAUNCH_KERNEL(3055); break;
	case 3056: LAUNCH_KERNEL(3056); break;
	case 3057: LAUNCH_KERNEL(3057); break;
	case 3058: LAUNCH_KERNEL(3058); break;
	case 3059: LAUNCH_KERNEL(3059); break;
	case 3060: LAUNCH_KERNEL(3060); break;
	case 3061: LAUNCH_KERNEL(3061); break;
	case 3062: LAUNCH_KERNEL(3062); break;
	case 3063: LAUNCH_KERNEL(3063); break;
	case 3064: LAUNCH_KERNEL(3064); break;
	case 3065: LAUNCH_KERNEL(3065); break;
	case 3066: LAUNCH_KERNEL(3066); break;
	case 3067: LAUNCH_KERNEL(3067); break;
	case 3068: LAUNCH_KERNEL(3068); break;
	case 3069: LAUNCH_KERNEL(3069); break;
	case 3070: LAUNCH_KERNEL(3070); break;
	case 3071: LAUNCH_KERNEL(3071); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
