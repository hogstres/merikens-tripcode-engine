// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 0
#include "CUDA10_Kernel.h"
#define SALT 1
#include "CUDA10_Kernel.h"
#define SALT 2
#include "CUDA10_Kernel.h"
#define SALT 3
#include "CUDA10_Kernel.h"
#define SALT 4
#include "CUDA10_Kernel.h"
#define SALT 5
#include "CUDA10_Kernel.h"
#define SALT 6
#include "CUDA10_Kernel.h"
#define SALT 7
#include "CUDA10_Kernel.h"
#define SALT 8
#include "CUDA10_Kernel.h"
#define SALT 9
#include "CUDA10_Kernel.h"
#define SALT 10
#include "CUDA10_Kernel.h"
#define SALT 11
#include "CUDA10_Kernel.h"
#define SALT 12
#include "CUDA10_Kernel.h"
#define SALT 13
#include "CUDA10_Kernel.h"
#define SALT 14
#include "CUDA10_Kernel.h"
#define SALT 15
#include "CUDA10_Kernel.h"
#define SALT 16
#include "CUDA10_Kernel.h"
#define SALT 17
#include "CUDA10_Kernel.h"
#define SALT 18
#include "CUDA10_Kernel.h"
#define SALT 19
#include "CUDA10_Kernel.h"
#define SALT 20
#include "CUDA10_Kernel.h"
#define SALT 21
#include "CUDA10_Kernel.h"
#define SALT 22
#include "CUDA10_Kernel.h"
#define SALT 23
#include "CUDA10_Kernel.h"
#define SALT 24
#include "CUDA10_Kernel.h"
#define SALT 25
#include "CUDA10_Kernel.h"
#define SALT 26
#include "CUDA10_Kernel.h"
#define SALT 27
#include "CUDA10_Kernel.h"
#define SALT 28
#include "CUDA10_Kernel.h"
#define SALT 29
#include "CUDA10_Kernel.h"
#define SALT 30
#include "CUDA10_Kernel.h"
#define SALT 31
#include "CUDA10_Kernel.h"
#define SALT 32
#include "CUDA10_Kernel.h"
#define SALT 33
#include "CUDA10_Kernel.h"
#define SALT 34
#include "CUDA10_Kernel.h"
#define SALT 35
#include "CUDA10_Kernel.h"
#define SALT 36
#include "CUDA10_Kernel.h"
#define SALT 37
#include "CUDA10_Kernel.h"
#define SALT 38
#include "CUDA10_Kernel.h"
#define SALT 39
#include "CUDA10_Kernel.h"
#define SALT 40
#include "CUDA10_Kernel.h"
#define SALT 41
#include "CUDA10_Kernel.h"
#define SALT 42
#include "CUDA10_Kernel.h"
#define SALT 43
#include "CUDA10_Kernel.h"
#define SALT 44
#include "CUDA10_Kernel.h"
#define SALT 45
#include "CUDA10_Kernel.h"
#define SALT 46
#include "CUDA10_Kernel.h"
#define SALT 47
#include "CUDA10_Kernel.h"
#define SALT 48
#include "CUDA10_Kernel.h"
#define SALT 49
#include "CUDA10_Kernel.h"
#define SALT 50
#include "CUDA10_Kernel.h"
#define SALT 51
#include "CUDA10_Kernel.h"
#define SALT 52
#include "CUDA10_Kernel.h"
#define SALT 53
#include "CUDA10_Kernel.h"
#define SALT 54
#include "CUDA10_Kernel.h"
#define SALT 55
#include "CUDA10_Kernel.h"
#define SALT 56
#include "CUDA10_Kernel.h"
#define SALT 57
#include "CUDA10_Kernel.h"
#define SALT 58
#include "CUDA10_Kernel.h"
#define SALT 59
#include "CUDA10_Kernel.h"
#define SALT 60
#include "CUDA10_Kernel.h"
#define SALT 61
#include "CUDA10_Kernel.h"
#define SALT 62
#include "CUDA10_Kernel.h"
#define SALT 63
#include "CUDA10_Kernel.h"
#define SALT 64
#include "CUDA10_Kernel.h"
#define SALT 65
#include "CUDA10_Kernel.h"
#define SALT 66
#include "CUDA10_Kernel.h"
#define SALT 67
#include "CUDA10_Kernel.h"
#define SALT 68
#include "CUDA10_Kernel.h"
#define SALT 69
#include "CUDA10_Kernel.h"
#define SALT 70
#include "CUDA10_Kernel.h"
#define SALT 71
#include "CUDA10_Kernel.h"
#define SALT 72
#include "CUDA10_Kernel.h"
#define SALT 73
#include "CUDA10_Kernel.h"
#define SALT 74
#include "CUDA10_Kernel.h"
#define SALT 75
#include "CUDA10_Kernel.h"
#define SALT 76
#include "CUDA10_Kernel.h"
#define SALT 77
#include "CUDA10_Kernel.h"
#define SALT 78
#include "CUDA10_Kernel.h"
#define SALT 79
#include "CUDA10_Kernel.h"
#define SALT 80
#include "CUDA10_Kernel.h"
#define SALT 81
#include "CUDA10_Kernel.h"
#define SALT 82
#include "CUDA10_Kernel.h"
#define SALT 83
#include "CUDA10_Kernel.h"
#define SALT 84
#include "CUDA10_Kernel.h"
#define SALT 85
#include "CUDA10_Kernel.h"
#define SALT 86
#include "CUDA10_Kernel.h"
#define SALT 87
#include "CUDA10_Kernel.h"
#define SALT 88
#include "CUDA10_Kernel.h"
#define SALT 89
#include "CUDA10_Kernel.h"
#define SALT 90
#include "CUDA10_Kernel.h"
#define SALT 91
#include "CUDA10_Kernel.h"
#define SALT 92
#include "CUDA10_Kernel.h"
#define SALT 93
#include "CUDA10_Kernel.h"
#define SALT 94
#include "CUDA10_Kernel.h"
#define SALT 95
#include "CUDA10_Kernel.h"
#define SALT 96
#include "CUDA10_Kernel.h"
#define SALT 97
#include "CUDA10_Kernel.h"
#define SALT 98
#include "CUDA10_Kernel.h"
#define SALT 99
#include "CUDA10_Kernel.h"
#define SALT 100
#include "CUDA10_Kernel.h"
#define SALT 101
#include "CUDA10_Kernel.h"
#define SALT 102
#include "CUDA10_Kernel.h"
#define SALT 103
#include "CUDA10_Kernel.h"
#define SALT 104
#include "CUDA10_Kernel.h"
#define SALT 105
#include "CUDA10_Kernel.h"
#define SALT 106
#include "CUDA10_Kernel.h"
#define SALT 107
#include "CUDA10_Kernel.h"
#define SALT 108
#include "CUDA10_Kernel.h"
#define SALT 109
#include "CUDA10_Kernel.h"
#define SALT 110
#include "CUDA10_Kernel.h"
#define SALT 111
#include "CUDA10_Kernel.h"
#define SALT 112
#include "CUDA10_Kernel.h"
#define SALT 113
#include "CUDA10_Kernel.h"
#define SALT 114
#include "CUDA10_Kernel.h"
#define SALT 115
#include "CUDA10_Kernel.h"
#define SALT 116
#include "CUDA10_Kernel.h"
#define SALT 117
#include "CUDA10_Kernel.h"
#define SALT 118
#include "CUDA10_Kernel.h"
#define SALT 119
#include "CUDA10_Kernel.h"
#define SALT 120
#include "CUDA10_Kernel.h"
#define SALT 121
#include "CUDA10_Kernel.h"
#define SALT 122
#include "CUDA10_Kernel.h"
#define SALT 123
#include "CUDA10_Kernel.h"
#define SALT 124
#include "CUDA10_Kernel.h"
#define SALT 125
#include "CUDA10_Kernel.h"
#define SALT 126
#include "CUDA10_Kernel.h"
#define SALT 127
#include "CUDA10_Kernel.h"
#define SALT 128
#include "CUDA10_Kernel.h"
#define SALT 129
#include "CUDA10_Kernel.h"
#define SALT 130
#include "CUDA10_Kernel.h"
#define SALT 131
#include "CUDA10_Kernel.h"
#define SALT 132
#include "CUDA10_Kernel.h"
#define SALT 133
#include "CUDA10_Kernel.h"
#define SALT 134
#include "CUDA10_Kernel.h"
#define SALT 135
#include "CUDA10_Kernel.h"
#define SALT 136
#include "CUDA10_Kernel.h"
#define SALT 137
#include "CUDA10_Kernel.h"
#define SALT 138
#include "CUDA10_Kernel.h"
#define SALT 139
#include "CUDA10_Kernel.h"
#define SALT 140
#include "CUDA10_Kernel.h"
#define SALT 141
#include "CUDA10_Kernel.h"
#define SALT 142
#include "CUDA10_Kernel.h"
#define SALT 143
#include "CUDA10_Kernel.h"
#define SALT 144
#include "CUDA10_Kernel.h"
#define SALT 145
#include "CUDA10_Kernel.h"
#define SALT 146
#include "CUDA10_Kernel.h"
#define SALT 147
#include "CUDA10_Kernel.h"
#define SALT 148
#include "CUDA10_Kernel.h"
#define SALT 149
#include "CUDA10_Kernel.h"
#define SALT 150
#include "CUDA10_Kernel.h"
#define SALT 151
#include "CUDA10_Kernel.h"
#define SALT 152
#include "CUDA10_Kernel.h"
#define SALT 153
#include "CUDA10_Kernel.h"
#define SALT 154
#include "CUDA10_Kernel.h"
#define SALT 155
#include "CUDA10_Kernel.h"
#define SALT 156
#include "CUDA10_Kernel.h"
#define SALT 157
#include "CUDA10_Kernel.h"
#define SALT 158
#include "CUDA10_Kernel.h"
#define SALT 159
#include "CUDA10_Kernel.h"
#define SALT 160
#include "CUDA10_Kernel.h"
#define SALT 161
#include "CUDA10_Kernel.h"
#define SALT 162
#include "CUDA10_Kernel.h"
#define SALT 163
#include "CUDA10_Kernel.h"
#define SALT 164
#include "CUDA10_Kernel.h"
#define SALT 165
#include "CUDA10_Kernel.h"
#define SALT 166
#include "CUDA10_Kernel.h"
#define SALT 167
#include "CUDA10_Kernel.h"
#define SALT 168
#include "CUDA10_Kernel.h"
#define SALT 169
#include "CUDA10_Kernel.h"
#define SALT 170
#include "CUDA10_Kernel.h"
#define SALT 171
#include "CUDA10_Kernel.h"
#define SALT 172
#include "CUDA10_Kernel.h"
#define SALT 173
#include "CUDA10_Kernel.h"
#define SALT 174
#include "CUDA10_Kernel.h"
#define SALT 175
#include "CUDA10_Kernel.h"
#define SALT 176
#include "CUDA10_Kernel.h"
#define SALT 177
#include "CUDA10_Kernel.h"
#define SALT 178
#include "CUDA10_Kernel.h"
#define SALT 179
#include "CUDA10_Kernel.h"
#define SALT 180
#include "CUDA10_Kernel.h"
#define SALT 181
#include "CUDA10_Kernel.h"
#define SALT 182
#include "CUDA10_Kernel.h"
#define SALT 183
#include "CUDA10_Kernel.h"
#define SALT 184
#include "CUDA10_Kernel.h"
#define SALT 185
#include "CUDA10_Kernel.h"
#define SALT 186
#include "CUDA10_Kernel.h"
#define SALT 187
#include "CUDA10_Kernel.h"
#define SALT 188
#include "CUDA10_Kernel.h"
#define SALT 189
#include "CUDA10_Kernel.h"
#define SALT 190
#include "CUDA10_Kernel.h"
#define SALT 191
#include "CUDA10_Kernel.h"
#define SALT 192
#include "CUDA10_Kernel.h"
#define SALT 193
#include "CUDA10_Kernel.h"
#define SALT 194
#include "CUDA10_Kernel.h"
#define SALT 195
#include "CUDA10_Kernel.h"
#define SALT 196
#include "CUDA10_Kernel.h"
#define SALT 197
#include "CUDA10_Kernel.h"
#define SALT 198
#include "CUDA10_Kernel.h"
#define SALT 199
#include "CUDA10_Kernel.h"
#define SALT 200
#include "CUDA10_Kernel.h"
#define SALT 201
#include "CUDA10_Kernel.h"
#define SALT 202
#include "CUDA10_Kernel.h"
#define SALT 203
#include "CUDA10_Kernel.h"
#define SALT 204
#include "CUDA10_Kernel.h"
#define SALT 205
#include "CUDA10_Kernel.h"
#define SALT 206
#include "CUDA10_Kernel.h"
#define SALT 207
#include "CUDA10_Kernel.h"
#define SALT 208
#include "CUDA10_Kernel.h"
#define SALT 209
#include "CUDA10_Kernel.h"
#define SALT 210
#include "CUDA10_Kernel.h"
#define SALT 211
#include "CUDA10_Kernel.h"
#define SALT 212
#include "CUDA10_Kernel.h"
#define SALT 213
#include "CUDA10_Kernel.h"
#define SALT 214
#include "CUDA10_Kernel.h"
#define SALT 215
#include "CUDA10_Kernel.h"
#define SALT 216
#include "CUDA10_Kernel.h"
#define SALT 217
#include "CUDA10_Kernel.h"
#define SALT 218
#include "CUDA10_Kernel.h"
#define SALT 219
#include "CUDA10_Kernel.h"
#define SALT 220
#include "CUDA10_Kernel.h"
#define SALT 221
#include "CUDA10_Kernel.h"
#define SALT 222
#include "CUDA10_Kernel.h"
#define SALT 223
#include "CUDA10_Kernel.h"
#define SALT 224
#include "CUDA10_Kernel.h"
#define SALT 225
#include "CUDA10_Kernel.h"
#define SALT 226
#include "CUDA10_Kernel.h"
#define SALT 227
#include "CUDA10_Kernel.h"
#define SALT 228
#include "CUDA10_Kernel.h"
#define SALT 229
#include "CUDA10_Kernel.h"
#define SALT 230
#include "CUDA10_Kernel.h"
#define SALT 231
#include "CUDA10_Kernel.h"
#define SALT 232
#include "CUDA10_Kernel.h"
#define SALT 233
#include "CUDA10_Kernel.h"
#define SALT 234
#include "CUDA10_Kernel.h"
#define SALT 235
#include "CUDA10_Kernel.h"
#define SALT 236
#include "CUDA10_Kernel.h"
#define SALT 237
#include "CUDA10_Kernel.h"
#define SALT 238
#include "CUDA10_Kernel.h"
#define SALT 239
#include "CUDA10_Kernel.h"
#define SALT 240
#include "CUDA10_Kernel.h"
#define SALT 241
#include "CUDA10_Kernel.h"
#define SALT 242
#include "CUDA10_Kernel.h"
#define SALT 243
#include "CUDA10_Kernel.h"
#define SALT 244
#include "CUDA10_Kernel.h"
#define SALT 245
#include "CUDA10_Kernel.h"
#define SALT 246
#include "CUDA10_Kernel.h"
#define SALT 247
#include "CUDA10_Kernel.h"
#define SALT 248
#include "CUDA10_Kernel.h"
#define SALT 249
#include "CUDA10_Kernel.h"
#define SALT 250
#include "CUDA10_Kernel.h"
#define SALT 251
#include "CUDA10_Kernel.h"
#define SALT 252
#include "CUDA10_Kernel.h"
#define SALT 253
#include "CUDA10_Kernel.h"
#define SALT 254
#include "CUDA10_Kernel.h"
#define SALT 255
#include "CUDA10_Kernel.h"
#define SALT 256
#include "CUDA10_Kernel.h"
#define SALT 257
#include "CUDA10_Kernel.h"
#define SALT 258
#include "CUDA10_Kernel.h"
#define SALT 259
#include "CUDA10_Kernel.h"
#define SALT 260
#include "CUDA10_Kernel.h"
#define SALT 261
#include "CUDA10_Kernel.h"
#define SALT 262
#include "CUDA10_Kernel.h"
#define SALT 263
#include "CUDA10_Kernel.h"
#define SALT 264
#include "CUDA10_Kernel.h"
#define SALT 265
#include "CUDA10_Kernel.h"
#define SALT 266
#include "CUDA10_Kernel.h"
#define SALT 267
#include "CUDA10_Kernel.h"
#define SALT 268
#include "CUDA10_Kernel.h"
#define SALT 269
#include "CUDA10_Kernel.h"
#define SALT 270
#include "CUDA10_Kernel.h"
#define SALT 271
#include "CUDA10_Kernel.h"
#define SALT 272
#include "CUDA10_Kernel.h"
#define SALT 273
#include "CUDA10_Kernel.h"
#define SALT 274
#include "CUDA10_Kernel.h"
#define SALT 275
#include "CUDA10_Kernel.h"
#define SALT 276
#include "CUDA10_Kernel.h"
#define SALT 277
#include "CUDA10_Kernel.h"
#define SALT 278
#include "CUDA10_Kernel.h"
#define SALT 279
#include "CUDA10_Kernel.h"
#define SALT 280
#include "CUDA10_Kernel.h"
#define SALT 281
#include "CUDA10_Kernel.h"
#define SALT 282
#include "CUDA10_Kernel.h"
#define SALT 283
#include "CUDA10_Kernel.h"
#define SALT 284
#include "CUDA10_Kernel.h"
#define SALT 285
#include "CUDA10_Kernel.h"
#define SALT 286
#include "CUDA10_Kernel.h"
#define SALT 287
#include "CUDA10_Kernel.h"
#define SALT 288
#include "CUDA10_Kernel.h"
#define SALT 289
#include "CUDA10_Kernel.h"
#define SALT 290
#include "CUDA10_Kernel.h"
#define SALT 291
#include "CUDA10_Kernel.h"
#define SALT 292
#include "CUDA10_Kernel.h"
#define SALT 293
#include "CUDA10_Kernel.h"
#define SALT 294
#include "CUDA10_Kernel.h"
#define SALT 295
#include "CUDA10_Kernel.h"
#define SALT 296
#include "CUDA10_Kernel.h"
#define SALT 297
#include "CUDA10_Kernel.h"
#define SALT 298
#include "CUDA10_Kernel.h"
#define SALT 299
#include "CUDA10_Kernel.h"
#define SALT 300
#include "CUDA10_Kernel.h"
#define SALT 301
#include "CUDA10_Kernel.h"
#define SALT 302
#include "CUDA10_Kernel.h"
#define SALT 303
#include "CUDA10_Kernel.h"
#define SALT 304
#include "CUDA10_Kernel.h"
#define SALT 305
#include "CUDA10_Kernel.h"
#define SALT 306
#include "CUDA10_Kernel.h"
#define SALT 307
#include "CUDA10_Kernel.h"
#define SALT 308
#include "CUDA10_Kernel.h"
#define SALT 309
#include "CUDA10_Kernel.h"
#define SALT 310
#include "CUDA10_Kernel.h"
#define SALT 311
#include "CUDA10_Kernel.h"
#define SALT 312
#include "CUDA10_Kernel.h"
#define SALT 313
#include "CUDA10_Kernel.h"
#define SALT 314
#include "CUDA10_Kernel.h"
#define SALT 315
#include "CUDA10_Kernel.h"
#define SALT 316
#include "CUDA10_Kernel.h"
#define SALT 317
#include "CUDA10_Kernel.h"
#define SALT 318
#include "CUDA10_Kernel.h"
#define SALT 319
#include "CUDA10_Kernel.h"
#define SALT 320
#include "CUDA10_Kernel.h"
#define SALT 321
#include "CUDA10_Kernel.h"
#define SALT 322
#include "CUDA10_Kernel.h"
#define SALT 323
#include "CUDA10_Kernel.h"
#define SALT 324
#include "CUDA10_Kernel.h"
#define SALT 325
#include "CUDA10_Kernel.h"
#define SALT 326
#include "CUDA10_Kernel.h"
#define SALT 327
#include "CUDA10_Kernel.h"
#define SALT 328
#include "CUDA10_Kernel.h"
#define SALT 329
#include "CUDA10_Kernel.h"
#define SALT 330
#include "CUDA10_Kernel.h"
#define SALT 331
#include "CUDA10_Kernel.h"
#define SALT 332
#include "CUDA10_Kernel.h"
#define SALT 333
#include "CUDA10_Kernel.h"
#define SALT 334
#include "CUDA10_Kernel.h"
#define SALT 335
#include "CUDA10_Kernel.h"
#define SALT 336
#include "CUDA10_Kernel.h"
#define SALT 337
#include "CUDA10_Kernel.h"
#define SALT 338
#include "CUDA10_Kernel.h"
#define SALT 339
#include "CUDA10_Kernel.h"
#define SALT 340
#include "CUDA10_Kernel.h"
#define SALT 341
#include "CUDA10_Kernel.h"
#define SALT 342
#include "CUDA10_Kernel.h"
#define SALT 343
#include "CUDA10_Kernel.h"
#define SALT 344
#include "CUDA10_Kernel.h"
#define SALT 345
#include "CUDA10_Kernel.h"
#define SALT 346
#include "CUDA10_Kernel.h"
#define SALT 347
#include "CUDA10_Kernel.h"
#define SALT 348
#include "CUDA10_Kernel.h"
#define SALT 349
#include "CUDA10_Kernel.h"
#define SALT 350
#include "CUDA10_Kernel.h"
#define SALT 351
#include "CUDA10_Kernel.h"
#define SALT 352
#include "CUDA10_Kernel.h"
#define SALT 353
#include "CUDA10_Kernel.h"
#define SALT 354
#include "CUDA10_Kernel.h"
#define SALT 355
#include "CUDA10_Kernel.h"
#define SALT 356
#include "CUDA10_Kernel.h"
#define SALT 357
#include "CUDA10_Kernel.h"
#define SALT 358
#include "CUDA10_Kernel.h"
#define SALT 359
#include "CUDA10_Kernel.h"
#define SALT 360
#include "CUDA10_Kernel.h"
#define SALT 361
#include "CUDA10_Kernel.h"
#define SALT 362
#include "CUDA10_Kernel.h"
#define SALT 363
#include "CUDA10_Kernel.h"
#define SALT 364
#include "CUDA10_Kernel.h"
#define SALT 365
#include "CUDA10_Kernel.h"
#define SALT 366
#include "CUDA10_Kernel.h"
#define SALT 367
#include "CUDA10_Kernel.h"
#define SALT 368
#include "CUDA10_Kernel.h"
#define SALT 369
#include "CUDA10_Kernel.h"
#define SALT 370
#include "CUDA10_Kernel.h"
#define SALT 371
#include "CUDA10_Kernel.h"
#define SALT 372
#include "CUDA10_Kernel.h"
#define SALT 373
#include "CUDA10_Kernel.h"
#define SALT 374
#include "CUDA10_Kernel.h"
#define SALT 375
#include "CUDA10_Kernel.h"
#define SALT 376
#include "CUDA10_Kernel.h"
#define SALT 377
#include "CUDA10_Kernel.h"
#define SALT 378
#include "CUDA10_Kernel.h"
#define SALT 379
#include "CUDA10_Kernel.h"
#define SALT 380
#include "CUDA10_Kernel.h"
#define SALT 381
#include "CUDA10_Kernel.h"
#define SALT 382
#include "CUDA10_Kernel.h"
#define SALT 383
#include "CUDA10_Kernel.h"
#define SALT 384
#include "CUDA10_Kernel.h"
#define SALT 385
#include "CUDA10_Kernel.h"
#define SALT 386
#include "CUDA10_Kernel.h"
#define SALT 387
#include "CUDA10_Kernel.h"
#define SALT 388
#include "CUDA10_Kernel.h"
#define SALT 389
#include "CUDA10_Kernel.h"
#define SALT 390
#include "CUDA10_Kernel.h"
#define SALT 391
#include "CUDA10_Kernel.h"
#define SALT 392
#include "CUDA10_Kernel.h"
#define SALT 393
#include "CUDA10_Kernel.h"
#define SALT 394
#include "CUDA10_Kernel.h"
#define SALT 395
#include "CUDA10_Kernel.h"
#define SALT 396
#include "CUDA10_Kernel.h"
#define SALT 397
#include "CUDA10_Kernel.h"
#define SALT 398
#include "CUDA10_Kernel.h"
#define SALT 399
#include "CUDA10_Kernel.h"
#define SALT 400
#include "CUDA10_Kernel.h"
#define SALT 401
#include "CUDA10_Kernel.h"
#define SALT 402
#include "CUDA10_Kernel.h"
#define SALT 403
#include "CUDA10_Kernel.h"
#define SALT 404
#include "CUDA10_Kernel.h"
#define SALT 405
#include "CUDA10_Kernel.h"
#define SALT 406
#include "CUDA10_Kernel.h"
#define SALT 407
#include "CUDA10_Kernel.h"
#define SALT 408
#include "CUDA10_Kernel.h"
#define SALT 409
#include "CUDA10_Kernel.h"
#define SALT 410
#include "CUDA10_Kernel.h"
#define SALT 411
#include "CUDA10_Kernel.h"
#define SALT 412
#include "CUDA10_Kernel.h"
#define SALT 413
#include "CUDA10_Kernel.h"
#define SALT 414
#include "CUDA10_Kernel.h"
#define SALT 415
#include "CUDA10_Kernel.h"
#define SALT 416
#include "CUDA10_Kernel.h"
#define SALT 417
#include "CUDA10_Kernel.h"
#define SALT 418
#include "CUDA10_Kernel.h"
#define SALT 419
#include "CUDA10_Kernel.h"
#define SALT 420
#include "CUDA10_Kernel.h"
#define SALT 421
#include "CUDA10_Kernel.h"
#define SALT 422
#include "CUDA10_Kernel.h"
#define SALT 423
#include "CUDA10_Kernel.h"
#define SALT 424
#include "CUDA10_Kernel.h"
#define SALT 425
#include "CUDA10_Kernel.h"
#define SALT 426
#include "CUDA10_Kernel.h"
#define SALT 427
#include "CUDA10_Kernel.h"
#define SALT 428
#include "CUDA10_Kernel.h"
#define SALT 429
#include "CUDA10_Kernel.h"
#define SALT 430
#include "CUDA10_Kernel.h"
#define SALT 431
#include "CUDA10_Kernel.h"
#define SALT 432
#include "CUDA10_Kernel.h"
#define SALT 433
#include "CUDA10_Kernel.h"
#define SALT 434
#include "CUDA10_Kernel.h"
#define SALT 435
#include "CUDA10_Kernel.h"
#define SALT 436
#include "CUDA10_Kernel.h"
#define SALT 437
#include "CUDA10_Kernel.h"
#define SALT 438
#include "CUDA10_Kernel.h"
#define SALT 439
#include "CUDA10_Kernel.h"
#define SALT 440
#include "CUDA10_Kernel.h"
#define SALT 441
#include "CUDA10_Kernel.h"
#define SALT 442
#include "CUDA10_Kernel.h"
#define SALT 443
#include "CUDA10_Kernel.h"
#define SALT 444
#include "CUDA10_Kernel.h"
#define SALT 445
#include "CUDA10_Kernel.h"
#define SALT 446
#include "CUDA10_Kernel.h"
#define SALT 447
#include "CUDA10_Kernel.h"
#define SALT 448
#include "CUDA10_Kernel.h"
#define SALT 449
#include "CUDA10_Kernel.h"
#define SALT 450
#include "CUDA10_Kernel.h"
#define SALT 451
#include "CUDA10_Kernel.h"
#define SALT 452
#include "CUDA10_Kernel.h"
#define SALT 453
#include "CUDA10_Kernel.h"
#define SALT 454
#include "CUDA10_Kernel.h"
#define SALT 455
#include "CUDA10_Kernel.h"
#define SALT 456
#include "CUDA10_Kernel.h"
#define SALT 457
#include "CUDA10_Kernel.h"
#define SALT 458
#include "CUDA10_Kernel.h"
#define SALT 459
#include "CUDA10_Kernel.h"
#define SALT 460
#include "CUDA10_Kernel.h"
#define SALT 461
#include "CUDA10_Kernel.h"
#define SALT 462
#include "CUDA10_Kernel.h"
#define SALT 463
#include "CUDA10_Kernel.h"
#define SALT 464
#include "CUDA10_Kernel.h"
#define SALT 465
#include "CUDA10_Kernel.h"
#define SALT 466
#include "CUDA10_Kernel.h"
#define SALT 467
#include "CUDA10_Kernel.h"
#define SALT 468
#include "CUDA10_Kernel.h"
#define SALT 469
#include "CUDA10_Kernel.h"
#define SALT 470
#include "CUDA10_Kernel.h"
#define SALT 471
#include "CUDA10_Kernel.h"
#define SALT 472
#include "CUDA10_Kernel.h"
#define SALT 473
#include "CUDA10_Kernel.h"
#define SALT 474
#include "CUDA10_Kernel.h"
#define SALT 475
#include "CUDA10_Kernel.h"
#define SALT 476
#include "CUDA10_Kernel.h"
#define SALT 477
#include "CUDA10_Kernel.h"
#define SALT 478
#include "CUDA10_Kernel.h"
#define SALT 479
#include "CUDA10_Kernel.h"
#define SALT 480
#include "CUDA10_Kernel.h"
#define SALT 481
#include "CUDA10_Kernel.h"
#define SALT 482
#include "CUDA10_Kernel.h"
#define SALT 483
#include "CUDA10_Kernel.h"
#define SALT 484
#include "CUDA10_Kernel.h"
#define SALT 485
#include "CUDA10_Kernel.h"
#define SALT 486
#include "CUDA10_Kernel.h"
#define SALT 487
#include "CUDA10_Kernel.h"
#define SALT 488
#include "CUDA10_Kernel.h"
#define SALT 489
#include "CUDA10_Kernel.h"
#define SALT 490
#include "CUDA10_Kernel.h"
#define SALT 491
#include "CUDA10_Kernel.h"
#define SALT 492
#include "CUDA10_Kernel.h"
#define SALT 493
#include "CUDA10_Kernel.h"
#define SALT 494
#include "CUDA10_Kernel.h"
#define SALT 495
#include "CUDA10_Kernel.h"
#define SALT 496
#include "CUDA10_Kernel.h"
#define SALT 497
#include "CUDA10_Kernel.h"
#define SALT 498
#include "CUDA10_Kernel.h"
#define SALT 499
#include "CUDA10_Kernel.h"
#define SALT 500
#include "CUDA10_Kernel.h"
#define SALT 501
#include "CUDA10_Kernel.h"
#define SALT 502
#include "CUDA10_Kernel.h"
#define SALT 503
#include "CUDA10_Kernel.h"
#define SALT 504
#include "CUDA10_Kernel.h"
#define SALT 505
#include "CUDA10_Kernel.h"
#define SALT 506
#include "CUDA10_Kernel.h"
#define SALT 507
#include "CUDA10_Kernel.h"
#define SALT 508
#include "CUDA10_Kernel.h"
#define SALT 509
#include "CUDA10_Kernel.h"
#define SALT 510
#include "CUDA10_Kernel.h"
#define SALT 511
#include "CUDA10_Kernel.h"



void CUDA_DES_InitializeKernelLauncher0()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactSmallKeyBitmap), compactSmallKeyBitmap, sizeof(unsigned char) * COMPACT_SMALL_KEY_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
}

void CUDA_DES_LaunchKernel0(
	dim3 dimGrid, 
	dim3 dimBlock,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned char *cudaKeyBitmap,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	switch (intSalt) {
		case 0: LAUNCH_KERNEL(0); break;
		case 1: LAUNCH_KERNEL(1); break;
		case 2: LAUNCH_KERNEL(2); break;
		case 3: LAUNCH_KERNEL(3); break;
		case 4: LAUNCH_KERNEL(4); break;
		case 5: LAUNCH_KERNEL(5); break;
		case 6: LAUNCH_KERNEL(6); break;
		case 7: LAUNCH_KERNEL(7); break;
		case 8: LAUNCH_KERNEL(8); break;
		case 9: LAUNCH_KERNEL(9); break;
		case 10: LAUNCH_KERNEL(10); break;
		case 11: LAUNCH_KERNEL(11); break;
		case 12: LAUNCH_KERNEL(12); break;
		case 13: LAUNCH_KERNEL(13); break;
		case 14: LAUNCH_KERNEL(14); break;
		case 15: LAUNCH_KERNEL(15); break;
		case 16: LAUNCH_KERNEL(16); break;
		case 17: LAUNCH_KERNEL(17); break;
		case 18: LAUNCH_KERNEL(18); break;
		case 19: LAUNCH_KERNEL(19); break;
		case 20: LAUNCH_KERNEL(20); break;
		case 21: LAUNCH_KERNEL(21); break;
		case 22: LAUNCH_KERNEL(22); break;
		case 23: LAUNCH_KERNEL(23); break;
		case 24: LAUNCH_KERNEL(24); break;
		case 25: LAUNCH_KERNEL(25); break;
		case 26: LAUNCH_KERNEL(26); break;
		case 27: LAUNCH_KERNEL(27); break;
		case 28: LAUNCH_KERNEL(28); break;
		case 29: LAUNCH_KERNEL(29); break;
		case 30: LAUNCH_KERNEL(30); break;
		case 31: LAUNCH_KERNEL(31); break;
		case 32: LAUNCH_KERNEL(32); break;
		case 33: LAUNCH_KERNEL(33); break;
		case 34: LAUNCH_KERNEL(34); break;
		case 35: LAUNCH_KERNEL(35); break;
		case 36: LAUNCH_KERNEL(36); break;
		case 37: LAUNCH_KERNEL(37); break;
		case 38: LAUNCH_KERNEL(38); break;
		case 39: LAUNCH_KERNEL(39); break;
		case 40: LAUNCH_KERNEL(40); break;
		case 41: LAUNCH_KERNEL(41); break;
		case 42: LAUNCH_KERNEL(42); break;
		case 43: LAUNCH_KERNEL(43); break;
		case 44: LAUNCH_KERNEL(44); break;
		case 45: LAUNCH_KERNEL(45); break;
		case 46: LAUNCH_KERNEL(46); break;
		case 47: LAUNCH_KERNEL(47); break;
		case 48: LAUNCH_KERNEL(48); break;
		case 49: LAUNCH_KERNEL(49); break;
		case 50: LAUNCH_KERNEL(50); break;
		case 51: LAUNCH_KERNEL(51); break;
		case 52: LAUNCH_KERNEL(52); break;
		case 53: LAUNCH_KERNEL(53); break;
		case 54: LAUNCH_KERNEL(54); break;
		case 55: LAUNCH_KERNEL(55); break;
		case 56: LAUNCH_KERNEL(56); break;
		case 57: LAUNCH_KERNEL(57); break;
		case 58: LAUNCH_KERNEL(58); break;
		case 59: LAUNCH_KERNEL(59); break;
		case 60: LAUNCH_KERNEL(60); break;
		case 61: LAUNCH_KERNEL(61); break;
		case 62: LAUNCH_KERNEL(62); break;
		case 63: LAUNCH_KERNEL(63); break;
		case 64: LAUNCH_KERNEL(64); break;
		case 65: LAUNCH_KERNEL(65); break;
		case 66: LAUNCH_KERNEL(66); break;
		case 67: LAUNCH_KERNEL(67); break;
		case 68: LAUNCH_KERNEL(68); break;
		case 69: LAUNCH_KERNEL(69); break;
		case 70: LAUNCH_KERNEL(70); break;
		case 71: LAUNCH_KERNEL(71); break;
		case 72: LAUNCH_KERNEL(72); break;
		case 73: LAUNCH_KERNEL(73); break;
		case 74: LAUNCH_KERNEL(74); break;
		case 75: LAUNCH_KERNEL(75); break;
		case 76: LAUNCH_KERNEL(76); break;
		case 77: LAUNCH_KERNEL(77); break;
		case 78: LAUNCH_KERNEL(78); break;
		case 79: LAUNCH_KERNEL(79); break;
		case 80: LAUNCH_KERNEL(80); break;
		case 81: LAUNCH_KERNEL(81); break;
		case 82: LAUNCH_KERNEL(82); break;
		case 83: LAUNCH_KERNEL(83); break;
		case 84: LAUNCH_KERNEL(84); break;
		case 85: LAUNCH_KERNEL(85); break;
		case 86: LAUNCH_KERNEL(86); break;
		case 87: LAUNCH_KERNEL(87); break;
		case 88: LAUNCH_KERNEL(88); break;
		case 89: LAUNCH_KERNEL(89); break;
		case 90: LAUNCH_KERNEL(90); break;
		case 91: LAUNCH_KERNEL(91); break;
		case 92: LAUNCH_KERNEL(92); break;
		case 93: LAUNCH_KERNEL(93); break;
		case 94: LAUNCH_KERNEL(94); break;
		case 95: LAUNCH_KERNEL(95); break;
		case 96: LAUNCH_KERNEL(96); break;
		case 97: LAUNCH_KERNEL(97); break;
		case 98: LAUNCH_KERNEL(98); break;
		case 99: LAUNCH_KERNEL(99); break;
		case 100: LAUNCH_KERNEL(100); break;
		case 101: LAUNCH_KERNEL(101); break;
		case 102: LAUNCH_KERNEL(102); break;
		case 103: LAUNCH_KERNEL(103); break;
		case 104: LAUNCH_KERNEL(104); break;
		case 105: LAUNCH_KERNEL(105); break;
		case 106: LAUNCH_KERNEL(106); break;
		case 107: LAUNCH_KERNEL(107); break;
		case 108: LAUNCH_KERNEL(108); break;
		case 109: LAUNCH_KERNEL(109); break;
		case 110: LAUNCH_KERNEL(110); break;
		case 111: LAUNCH_KERNEL(111); break;
		case 112: LAUNCH_KERNEL(112); break;
		case 113: LAUNCH_KERNEL(113); break;
		case 114: LAUNCH_KERNEL(114); break;
		case 115: LAUNCH_KERNEL(115); break;
		case 116: LAUNCH_KERNEL(116); break;
		case 117: LAUNCH_KERNEL(117); break;
		case 118: LAUNCH_KERNEL(118); break;
		case 119: LAUNCH_KERNEL(119); break;
		case 120: LAUNCH_KERNEL(120); break;
		case 121: LAUNCH_KERNEL(121); break;
		case 122: LAUNCH_KERNEL(122); break;
		case 123: LAUNCH_KERNEL(123); break;
		case 124: LAUNCH_KERNEL(124); break;
		case 125: LAUNCH_KERNEL(125); break;
		case 126: LAUNCH_KERNEL(126); break;
		case 127: LAUNCH_KERNEL(127); break;
		case 128: LAUNCH_KERNEL(128); break;
		case 129: LAUNCH_KERNEL(129); break;
		case 130: LAUNCH_KERNEL(130); break;
		case 131: LAUNCH_KERNEL(131); break;
		case 132: LAUNCH_KERNEL(132); break;
		case 133: LAUNCH_KERNEL(133); break;
		case 134: LAUNCH_KERNEL(134); break;
		case 135: LAUNCH_KERNEL(135); break;
		case 136: LAUNCH_KERNEL(136); break;
		case 137: LAUNCH_KERNEL(137); break;
		case 138: LAUNCH_KERNEL(138); break;
		case 139: LAUNCH_KERNEL(139); break;
		case 140: LAUNCH_KERNEL(140); break;
		case 141: LAUNCH_KERNEL(141); break;
		case 142: LAUNCH_KERNEL(142); break;
		case 143: LAUNCH_KERNEL(143); break;
		case 144: LAUNCH_KERNEL(144); break;
		case 145: LAUNCH_KERNEL(145); break;
		case 146: LAUNCH_KERNEL(146); break;
		case 147: LAUNCH_KERNEL(147); break;
		case 148: LAUNCH_KERNEL(148); break;
		case 149: LAUNCH_KERNEL(149); break;
		case 150: LAUNCH_KERNEL(150); break;
		case 151: LAUNCH_KERNEL(151); break;
		case 152: LAUNCH_KERNEL(152); break;
		case 153: LAUNCH_KERNEL(153); break;
		case 154: LAUNCH_KERNEL(154); break;
		case 155: LAUNCH_KERNEL(155); break;
		case 156: LAUNCH_KERNEL(156); break;
		case 157: LAUNCH_KERNEL(157); break;
		case 158: LAUNCH_KERNEL(158); break;
		case 159: LAUNCH_KERNEL(159); break;
		case 160: LAUNCH_KERNEL(160); break;
		case 161: LAUNCH_KERNEL(161); break;
		case 162: LAUNCH_KERNEL(162); break;
		case 163: LAUNCH_KERNEL(163); break;
		case 164: LAUNCH_KERNEL(164); break;
		case 165: LAUNCH_KERNEL(165); break;
		case 166: LAUNCH_KERNEL(166); break;
		case 167: LAUNCH_KERNEL(167); break;
		case 168: LAUNCH_KERNEL(168); break;
		case 169: LAUNCH_KERNEL(169); break;
		case 170: LAUNCH_KERNEL(170); break;
		case 171: LAUNCH_KERNEL(171); break;
		case 172: LAUNCH_KERNEL(172); break;
		case 173: LAUNCH_KERNEL(173); break;
		case 174: LAUNCH_KERNEL(174); break;
		case 175: LAUNCH_KERNEL(175); break;
		case 176: LAUNCH_KERNEL(176); break;
		case 177: LAUNCH_KERNEL(177); break;
		case 178: LAUNCH_KERNEL(178); break;
		case 179: LAUNCH_KERNEL(179); break;
		case 180: LAUNCH_KERNEL(180); break;
		case 181: LAUNCH_KERNEL(181); break;
		case 182: LAUNCH_KERNEL(182); break;
		case 183: LAUNCH_KERNEL(183); break;
		case 184: LAUNCH_KERNEL(184); break;
		case 185: LAUNCH_KERNEL(185); break;
		case 186: LAUNCH_KERNEL(186); break;
		case 187: LAUNCH_KERNEL(187); break;
		case 188: LAUNCH_KERNEL(188); break;
		case 189: LAUNCH_KERNEL(189); break;
		case 190: LAUNCH_KERNEL(190); break;
		case 191: LAUNCH_KERNEL(191); break;
		case 192: LAUNCH_KERNEL(192); break;
		case 193: LAUNCH_KERNEL(193); break;
		case 194: LAUNCH_KERNEL(194); break;
		case 195: LAUNCH_KERNEL(195); break;
		case 196: LAUNCH_KERNEL(196); break;
		case 197: LAUNCH_KERNEL(197); break;
		case 198: LAUNCH_KERNEL(198); break;
		case 199: LAUNCH_KERNEL(199); break;
		case 200: LAUNCH_KERNEL(200); break;
		case 201: LAUNCH_KERNEL(201); break;
		case 202: LAUNCH_KERNEL(202); break;
		case 203: LAUNCH_KERNEL(203); break;
		case 204: LAUNCH_KERNEL(204); break;
		case 205: LAUNCH_KERNEL(205); break;
		case 206: LAUNCH_KERNEL(206); break;
		case 207: LAUNCH_KERNEL(207); break;
		case 208: LAUNCH_KERNEL(208); break;
		case 209: LAUNCH_KERNEL(209); break;
		case 210: LAUNCH_KERNEL(210); break;
		case 211: LAUNCH_KERNEL(211); break;
		case 212: LAUNCH_KERNEL(212); break;
		case 213: LAUNCH_KERNEL(213); break;
		case 214: LAUNCH_KERNEL(214); break;
		case 215: LAUNCH_KERNEL(215); break;
		case 216: LAUNCH_KERNEL(216); break;
		case 217: LAUNCH_KERNEL(217); break;
		case 218: LAUNCH_KERNEL(218); break;
		case 219: LAUNCH_KERNEL(219); break;
		case 220: LAUNCH_KERNEL(220); break;
		case 221: LAUNCH_KERNEL(221); break;
		case 222: LAUNCH_KERNEL(222); break;
		case 223: LAUNCH_KERNEL(223); break;
		case 224: LAUNCH_KERNEL(224); break;
		case 225: LAUNCH_KERNEL(225); break;
		case 226: LAUNCH_KERNEL(226); break;
		case 227: LAUNCH_KERNEL(227); break;
		case 228: LAUNCH_KERNEL(228); break;
		case 229: LAUNCH_KERNEL(229); break;
		case 230: LAUNCH_KERNEL(230); break;
		case 231: LAUNCH_KERNEL(231); break;
		case 232: LAUNCH_KERNEL(232); break;
		case 233: LAUNCH_KERNEL(233); break;
		case 234: LAUNCH_KERNEL(234); break;
		case 235: LAUNCH_KERNEL(235); break;
		case 236: LAUNCH_KERNEL(236); break;
		case 237: LAUNCH_KERNEL(237); break;
		case 238: LAUNCH_KERNEL(238); break;
		case 239: LAUNCH_KERNEL(239); break;
		case 240: LAUNCH_KERNEL(240); break;
		case 241: LAUNCH_KERNEL(241); break;
		case 242: LAUNCH_KERNEL(242); break;
		case 243: LAUNCH_KERNEL(243); break;
		case 244: LAUNCH_KERNEL(244); break;
		case 245: LAUNCH_KERNEL(245); break;
		case 246: LAUNCH_KERNEL(246); break;
		case 247: LAUNCH_KERNEL(247); break;
		case 248: LAUNCH_KERNEL(248); break;
		case 249: LAUNCH_KERNEL(249); break;
		case 250: LAUNCH_KERNEL(250); break;
		case 251: LAUNCH_KERNEL(251); break;
		case 252: LAUNCH_KERNEL(252); break;
		case 253: LAUNCH_KERNEL(253); break;
		case 254: LAUNCH_KERNEL(254); break;
		case 255: LAUNCH_KERNEL(255); break;
		case 256: LAUNCH_KERNEL(256); break;
		case 257: LAUNCH_KERNEL(257); break;
		case 258: LAUNCH_KERNEL(258); break;
		case 259: LAUNCH_KERNEL(259); break;
		case 260: LAUNCH_KERNEL(260); break;
		case 261: LAUNCH_KERNEL(261); break;
		case 262: LAUNCH_KERNEL(262); break;
		case 263: LAUNCH_KERNEL(263); break;
		case 264: LAUNCH_KERNEL(264); break;
		case 265: LAUNCH_KERNEL(265); break;
		case 266: LAUNCH_KERNEL(266); break;
		case 267: LAUNCH_KERNEL(267); break;
		case 268: LAUNCH_KERNEL(268); break;
		case 269: LAUNCH_KERNEL(269); break;
		case 270: LAUNCH_KERNEL(270); break;
		case 271: LAUNCH_KERNEL(271); break;
		case 272: LAUNCH_KERNEL(272); break;
		case 273: LAUNCH_KERNEL(273); break;
		case 274: LAUNCH_KERNEL(274); break;
		case 275: LAUNCH_KERNEL(275); break;
		case 276: LAUNCH_KERNEL(276); break;
		case 277: LAUNCH_KERNEL(277); break;
		case 278: LAUNCH_KERNEL(278); break;
		case 279: LAUNCH_KERNEL(279); break;
		case 280: LAUNCH_KERNEL(280); break;
		case 281: LAUNCH_KERNEL(281); break;
		case 282: LAUNCH_KERNEL(282); break;
		case 283: LAUNCH_KERNEL(283); break;
		case 284: LAUNCH_KERNEL(284); break;
		case 285: LAUNCH_KERNEL(285); break;
		case 286: LAUNCH_KERNEL(286); break;
		case 287: LAUNCH_KERNEL(287); break;
		case 288: LAUNCH_KERNEL(288); break;
		case 289: LAUNCH_KERNEL(289); break;
		case 290: LAUNCH_KERNEL(290); break;
		case 291: LAUNCH_KERNEL(291); break;
		case 292: LAUNCH_KERNEL(292); break;
		case 293: LAUNCH_KERNEL(293); break;
		case 294: LAUNCH_KERNEL(294); break;
		case 295: LAUNCH_KERNEL(295); break;
		case 296: LAUNCH_KERNEL(296); break;
		case 297: LAUNCH_KERNEL(297); break;
		case 298: LAUNCH_KERNEL(298); break;
		case 299: LAUNCH_KERNEL(299); break;
		case 300: LAUNCH_KERNEL(300); break;
		case 301: LAUNCH_KERNEL(301); break;
		case 302: LAUNCH_KERNEL(302); break;
		case 303: LAUNCH_KERNEL(303); break;
		case 304: LAUNCH_KERNEL(304); break;
		case 305: LAUNCH_KERNEL(305); break;
		case 306: LAUNCH_KERNEL(306); break;
		case 307: LAUNCH_KERNEL(307); break;
		case 308: LAUNCH_KERNEL(308); break;
		case 309: LAUNCH_KERNEL(309); break;
		case 310: LAUNCH_KERNEL(310); break;
		case 311: LAUNCH_KERNEL(311); break;
		case 312: LAUNCH_KERNEL(312); break;
		case 313: LAUNCH_KERNEL(313); break;
		case 314: LAUNCH_KERNEL(314); break;
		case 315: LAUNCH_KERNEL(315); break;
		case 316: LAUNCH_KERNEL(316); break;
		case 317: LAUNCH_KERNEL(317); break;
		case 318: LAUNCH_KERNEL(318); break;
		case 319: LAUNCH_KERNEL(319); break;
		case 320: LAUNCH_KERNEL(320); break;
		case 321: LAUNCH_KERNEL(321); break;
		case 322: LAUNCH_KERNEL(322); break;
		case 323: LAUNCH_KERNEL(323); break;
		case 324: LAUNCH_KERNEL(324); break;
		case 325: LAUNCH_KERNEL(325); break;
		case 326: LAUNCH_KERNEL(326); break;
		case 327: LAUNCH_KERNEL(327); break;
		case 328: LAUNCH_KERNEL(328); break;
		case 329: LAUNCH_KERNEL(329); break;
		case 330: LAUNCH_KERNEL(330); break;
		case 331: LAUNCH_KERNEL(331); break;
		case 332: LAUNCH_KERNEL(332); break;
		case 333: LAUNCH_KERNEL(333); break;
		case 334: LAUNCH_KERNEL(334); break;
		case 335: LAUNCH_KERNEL(335); break;
		case 336: LAUNCH_KERNEL(336); break;
		case 337: LAUNCH_KERNEL(337); break;
		case 338: LAUNCH_KERNEL(338); break;
		case 339: LAUNCH_KERNEL(339); break;
		case 340: LAUNCH_KERNEL(340); break;
		case 341: LAUNCH_KERNEL(341); break;
		case 342: LAUNCH_KERNEL(342); break;
		case 343: LAUNCH_KERNEL(343); break;
		case 344: LAUNCH_KERNEL(344); break;
		case 345: LAUNCH_KERNEL(345); break;
		case 346: LAUNCH_KERNEL(346); break;
		case 347: LAUNCH_KERNEL(347); break;
		case 348: LAUNCH_KERNEL(348); break;
		case 349: LAUNCH_KERNEL(349); break;
		case 350: LAUNCH_KERNEL(350); break;
		case 351: LAUNCH_KERNEL(351); break;
		case 352: LAUNCH_KERNEL(352); break;
		case 353: LAUNCH_KERNEL(353); break;
		case 354: LAUNCH_KERNEL(354); break;
		case 355: LAUNCH_KERNEL(355); break;
		case 356: LAUNCH_KERNEL(356); break;
		case 357: LAUNCH_KERNEL(357); break;
		case 358: LAUNCH_KERNEL(358); break;
		case 359: LAUNCH_KERNEL(359); break;
		case 360: LAUNCH_KERNEL(360); break;
		case 361: LAUNCH_KERNEL(361); break;
		case 362: LAUNCH_KERNEL(362); break;
		case 363: LAUNCH_KERNEL(363); break;
		case 364: LAUNCH_KERNEL(364); break;
		case 365: LAUNCH_KERNEL(365); break;
		case 366: LAUNCH_KERNEL(366); break;
		case 367: LAUNCH_KERNEL(367); break;
		case 368: LAUNCH_KERNEL(368); break;
		case 369: LAUNCH_KERNEL(369); break;
		case 370: LAUNCH_KERNEL(370); break;
		case 371: LAUNCH_KERNEL(371); break;
		case 372: LAUNCH_KERNEL(372); break;
		case 373: LAUNCH_KERNEL(373); break;
		case 374: LAUNCH_KERNEL(374); break;
		case 375: LAUNCH_KERNEL(375); break;
		case 376: LAUNCH_KERNEL(376); break;
		case 377: LAUNCH_KERNEL(377); break;
		case 378: LAUNCH_KERNEL(378); break;
		case 379: LAUNCH_KERNEL(379); break;
		case 380: LAUNCH_KERNEL(380); break;
		case 381: LAUNCH_KERNEL(381); break;
		case 382: LAUNCH_KERNEL(382); break;
		case 383: LAUNCH_KERNEL(383); break;
		case 384: LAUNCH_KERNEL(384); break;
		case 385: LAUNCH_KERNEL(385); break;
		case 386: LAUNCH_KERNEL(386); break;
		case 387: LAUNCH_KERNEL(387); break;
		case 388: LAUNCH_KERNEL(388); break;
		case 389: LAUNCH_KERNEL(389); break;
		case 390: LAUNCH_KERNEL(390); break;
		case 391: LAUNCH_KERNEL(391); break;
		case 392: LAUNCH_KERNEL(392); break;
		case 393: LAUNCH_KERNEL(393); break;
		case 394: LAUNCH_KERNEL(394); break;
		case 395: LAUNCH_KERNEL(395); break;
		case 396: LAUNCH_KERNEL(396); break;
		case 397: LAUNCH_KERNEL(397); break;
		case 398: LAUNCH_KERNEL(398); break;
		case 399: LAUNCH_KERNEL(399); break;
		case 400: LAUNCH_KERNEL(400); break;
		case 401: LAUNCH_KERNEL(401); break;
		case 402: LAUNCH_KERNEL(402); break;
		case 403: LAUNCH_KERNEL(403); break;
		case 404: LAUNCH_KERNEL(404); break;
		case 405: LAUNCH_KERNEL(405); break;
		case 406: LAUNCH_KERNEL(406); break;
		case 407: LAUNCH_KERNEL(407); break;
		case 408: LAUNCH_KERNEL(408); break;
		case 409: LAUNCH_KERNEL(409); break;
		case 410: LAUNCH_KERNEL(410); break;
		case 411: LAUNCH_KERNEL(411); break;
		case 412: LAUNCH_KERNEL(412); break;
		case 413: LAUNCH_KERNEL(413); break;
		case 414: LAUNCH_KERNEL(414); break;
		case 415: LAUNCH_KERNEL(415); break;
		case 416: LAUNCH_KERNEL(416); break;
		case 417: LAUNCH_KERNEL(417); break;
		case 418: LAUNCH_KERNEL(418); break;
		case 419: LAUNCH_KERNEL(419); break;
		case 420: LAUNCH_KERNEL(420); break;
		case 421: LAUNCH_KERNEL(421); break;
		case 422: LAUNCH_KERNEL(422); break;
		case 423: LAUNCH_KERNEL(423); break;
		case 424: LAUNCH_KERNEL(424); break;
		case 425: LAUNCH_KERNEL(425); break;
		case 426: LAUNCH_KERNEL(426); break;
		case 427: LAUNCH_KERNEL(427); break;
		case 428: LAUNCH_KERNEL(428); break;
		case 429: LAUNCH_KERNEL(429); break;
		case 430: LAUNCH_KERNEL(430); break;
		case 431: LAUNCH_KERNEL(431); break;
		case 432: LAUNCH_KERNEL(432); break;
		case 433: LAUNCH_KERNEL(433); break;
		case 434: LAUNCH_KERNEL(434); break;
		case 435: LAUNCH_KERNEL(435); break;
		case 436: LAUNCH_KERNEL(436); break;
		case 437: LAUNCH_KERNEL(437); break;
		case 438: LAUNCH_KERNEL(438); break;
		case 439: LAUNCH_KERNEL(439); break;
		case 440: LAUNCH_KERNEL(440); break;
		case 441: LAUNCH_KERNEL(441); break;
		case 442: LAUNCH_KERNEL(442); break;
		case 443: LAUNCH_KERNEL(443); break;
		case 444: LAUNCH_KERNEL(444); break;
		case 445: LAUNCH_KERNEL(445); break;
		case 446: LAUNCH_KERNEL(446); break;
		case 447: LAUNCH_KERNEL(447); break;
		case 448: LAUNCH_KERNEL(448); break;
		case 449: LAUNCH_KERNEL(449); break;
		case 450: LAUNCH_KERNEL(450); break;
		case 451: LAUNCH_KERNEL(451); break;
		case 452: LAUNCH_KERNEL(452); break;
		case 453: LAUNCH_KERNEL(453); break;
		case 454: LAUNCH_KERNEL(454); break;
		case 455: LAUNCH_KERNEL(455); break;
		case 456: LAUNCH_KERNEL(456); break;
		case 457: LAUNCH_KERNEL(457); break;
		case 458: LAUNCH_KERNEL(458); break;
		case 459: LAUNCH_KERNEL(459); break;
		case 460: LAUNCH_KERNEL(460); break;
		case 461: LAUNCH_KERNEL(461); break;
		case 462: LAUNCH_KERNEL(462); break;
		case 463: LAUNCH_KERNEL(463); break;
		case 464: LAUNCH_KERNEL(464); break;
		case 465: LAUNCH_KERNEL(465); break;
		case 466: LAUNCH_KERNEL(466); break;
		case 467: LAUNCH_KERNEL(467); break;
		case 468: LAUNCH_KERNEL(468); break;
		case 469: LAUNCH_KERNEL(469); break;
		case 470: LAUNCH_KERNEL(470); break;
		case 471: LAUNCH_KERNEL(471); break;
		case 472: LAUNCH_KERNEL(472); break;
		case 473: LAUNCH_KERNEL(473); break;
		case 474: LAUNCH_KERNEL(474); break;
		case 475: LAUNCH_KERNEL(475); break;
		case 476: LAUNCH_KERNEL(476); break;
		case 477: LAUNCH_KERNEL(477); break;
		case 478: LAUNCH_KERNEL(478); break;
		case 479: LAUNCH_KERNEL(479); break;
		case 480: LAUNCH_KERNEL(480); break;
		case 481: LAUNCH_KERNEL(481); break;
		case 482: LAUNCH_KERNEL(482); break;
		case 483: LAUNCH_KERNEL(483); break;
		case 484: LAUNCH_KERNEL(484); break;
		case 485: LAUNCH_KERNEL(485); break;
		case 486: LAUNCH_KERNEL(486); break;
		case 487: LAUNCH_KERNEL(487); break;
		case 488: LAUNCH_KERNEL(488); break;
		case 489: LAUNCH_KERNEL(489); break;
		case 490: LAUNCH_KERNEL(490); break;
		case 491: LAUNCH_KERNEL(491); break;
		case 492: LAUNCH_KERNEL(492); break;
		case 493: LAUNCH_KERNEL(493); break;
		case 494: LAUNCH_KERNEL(494); break;
		case 495: LAUNCH_KERNEL(495); break;
		case 496: LAUNCH_KERNEL(496); break;
		case 497: LAUNCH_KERNEL(497); break;
		case 498: LAUNCH_KERNEL(498); break;
		case 499: LAUNCH_KERNEL(499); break;
		case 500: LAUNCH_KERNEL(500); break;
		case 501: LAUNCH_KERNEL(501); break;
		case 502: LAUNCH_KERNEL(502); break;
		case 503: LAUNCH_KERNEL(503); break;
		case 504: LAUNCH_KERNEL(504); break;
		case 505: LAUNCH_KERNEL(505); break;
		case 506: LAUNCH_KERNEL(506); break;
		case 507: LAUNCH_KERNEL(507); break;
		case 508: LAUNCH_KERNEL(508); break;
		case 509: LAUNCH_KERNEL(509); break;
		case 510: LAUNCH_KERNEL(510); break;
		case 511: LAUNCH_KERNEL(511); break;
		default: ASSERT(FALSE);
	}
}
