// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 1024
#include "CUDA10_Registers_Kernel.h"
#define SALT 1025
#include "CUDA10_Registers_Kernel.h"
#define SALT 1026
#include "CUDA10_Registers_Kernel.h"
#define SALT 1027
#include "CUDA10_Registers_Kernel.h"
#define SALT 1028
#include "CUDA10_Registers_Kernel.h"
#define SALT 1029
#include "CUDA10_Registers_Kernel.h"
#define SALT 1030
#include "CUDA10_Registers_Kernel.h"
#define SALT 1031
#include "CUDA10_Registers_Kernel.h"
#define SALT 1032
#include "CUDA10_Registers_Kernel.h"
#define SALT 1033
#include "CUDA10_Registers_Kernel.h"
#define SALT 1034
#include "CUDA10_Registers_Kernel.h"
#define SALT 1035
#include "CUDA10_Registers_Kernel.h"
#define SALT 1036
#include "CUDA10_Registers_Kernel.h"
#define SALT 1037
#include "CUDA10_Registers_Kernel.h"
#define SALT 1038
#include "CUDA10_Registers_Kernel.h"
#define SALT 1039
#include "CUDA10_Registers_Kernel.h"
#define SALT 1040
#include "CUDA10_Registers_Kernel.h"
#define SALT 1041
#include "CUDA10_Registers_Kernel.h"
#define SALT 1042
#include "CUDA10_Registers_Kernel.h"
#define SALT 1043
#include "CUDA10_Registers_Kernel.h"
#define SALT 1044
#include "CUDA10_Registers_Kernel.h"
#define SALT 1045
#include "CUDA10_Registers_Kernel.h"
#define SALT 1046
#include "CUDA10_Registers_Kernel.h"
#define SALT 1047
#include "CUDA10_Registers_Kernel.h"
#define SALT 1048
#include "CUDA10_Registers_Kernel.h"
#define SALT 1049
#include "CUDA10_Registers_Kernel.h"
#define SALT 1050
#include "CUDA10_Registers_Kernel.h"
#define SALT 1051
#include "CUDA10_Registers_Kernel.h"
#define SALT 1052
#include "CUDA10_Registers_Kernel.h"
#define SALT 1053
#include "CUDA10_Registers_Kernel.h"
#define SALT 1054
#include "CUDA10_Registers_Kernel.h"
#define SALT 1055
#include "CUDA10_Registers_Kernel.h"
#define SALT 1056
#include "CUDA10_Registers_Kernel.h"
#define SALT 1057
#include "CUDA10_Registers_Kernel.h"
#define SALT 1058
#include "CUDA10_Registers_Kernel.h"
#define SALT 1059
#include "CUDA10_Registers_Kernel.h"
#define SALT 1060
#include "CUDA10_Registers_Kernel.h"
#define SALT 1061
#include "CUDA10_Registers_Kernel.h"
#define SALT 1062
#include "CUDA10_Registers_Kernel.h"
#define SALT 1063
#include "CUDA10_Registers_Kernel.h"
#define SALT 1064
#include "CUDA10_Registers_Kernel.h"
#define SALT 1065
#include "CUDA10_Registers_Kernel.h"
#define SALT 1066
#include "CUDA10_Registers_Kernel.h"
#define SALT 1067
#include "CUDA10_Registers_Kernel.h"
#define SALT 1068
#include "CUDA10_Registers_Kernel.h"
#define SALT 1069
#include "CUDA10_Registers_Kernel.h"
#define SALT 1070
#include "CUDA10_Registers_Kernel.h"
#define SALT 1071
#include "CUDA10_Registers_Kernel.h"
#define SALT 1072
#include "CUDA10_Registers_Kernel.h"
#define SALT 1073
#include "CUDA10_Registers_Kernel.h"
#define SALT 1074
#include "CUDA10_Registers_Kernel.h"
#define SALT 1075
#include "CUDA10_Registers_Kernel.h"
#define SALT 1076
#include "CUDA10_Registers_Kernel.h"
#define SALT 1077
#include "CUDA10_Registers_Kernel.h"
#define SALT 1078
#include "CUDA10_Registers_Kernel.h"
#define SALT 1079
#include "CUDA10_Registers_Kernel.h"
#define SALT 1080
#include "CUDA10_Registers_Kernel.h"
#define SALT 1081
#include "CUDA10_Registers_Kernel.h"
#define SALT 1082
#include "CUDA10_Registers_Kernel.h"
#define SALT 1083
#include "CUDA10_Registers_Kernel.h"
#define SALT 1084
#include "CUDA10_Registers_Kernel.h"
#define SALT 1085
#include "CUDA10_Registers_Kernel.h"
#define SALT 1086
#include "CUDA10_Registers_Kernel.h"
#define SALT 1087
#include "CUDA10_Registers_Kernel.h"
#define SALT 1088
#include "CUDA10_Registers_Kernel.h"
#define SALT 1089
#include "CUDA10_Registers_Kernel.h"
#define SALT 1090
#include "CUDA10_Registers_Kernel.h"
#define SALT 1091
#include "CUDA10_Registers_Kernel.h"
#define SALT 1092
#include "CUDA10_Registers_Kernel.h"
#define SALT 1093
#include "CUDA10_Registers_Kernel.h"
#define SALT 1094
#include "CUDA10_Registers_Kernel.h"
#define SALT 1095
#include "CUDA10_Registers_Kernel.h"
#define SALT 1096
#include "CUDA10_Registers_Kernel.h"
#define SALT 1097
#include "CUDA10_Registers_Kernel.h"
#define SALT 1098
#include "CUDA10_Registers_Kernel.h"
#define SALT 1099
#include "CUDA10_Registers_Kernel.h"
#define SALT 1100
#include "CUDA10_Registers_Kernel.h"
#define SALT 1101
#include "CUDA10_Registers_Kernel.h"
#define SALT 1102
#include "CUDA10_Registers_Kernel.h"
#define SALT 1103
#include "CUDA10_Registers_Kernel.h"
#define SALT 1104
#include "CUDA10_Registers_Kernel.h"
#define SALT 1105
#include "CUDA10_Registers_Kernel.h"
#define SALT 1106
#include "CUDA10_Registers_Kernel.h"
#define SALT 1107
#include "CUDA10_Registers_Kernel.h"
#define SALT 1108
#include "CUDA10_Registers_Kernel.h"
#define SALT 1109
#include "CUDA10_Registers_Kernel.h"
#define SALT 1110
#include "CUDA10_Registers_Kernel.h"
#define SALT 1111
#include "CUDA10_Registers_Kernel.h"
#define SALT 1112
#include "CUDA10_Registers_Kernel.h"
#define SALT 1113
#include "CUDA10_Registers_Kernel.h"
#define SALT 1114
#include "CUDA10_Registers_Kernel.h"
#define SALT 1115
#include "CUDA10_Registers_Kernel.h"
#define SALT 1116
#include "CUDA10_Registers_Kernel.h"
#define SALT 1117
#include "CUDA10_Registers_Kernel.h"
#define SALT 1118
#include "CUDA10_Registers_Kernel.h"
#define SALT 1119
#include "CUDA10_Registers_Kernel.h"
#define SALT 1120
#include "CUDA10_Registers_Kernel.h"
#define SALT 1121
#include "CUDA10_Registers_Kernel.h"
#define SALT 1122
#include "CUDA10_Registers_Kernel.h"
#define SALT 1123
#include "CUDA10_Registers_Kernel.h"
#define SALT 1124
#include "CUDA10_Registers_Kernel.h"
#define SALT 1125
#include "CUDA10_Registers_Kernel.h"
#define SALT 1126
#include "CUDA10_Registers_Kernel.h"
#define SALT 1127
#include "CUDA10_Registers_Kernel.h"
#define SALT 1128
#include "CUDA10_Registers_Kernel.h"
#define SALT 1129
#include "CUDA10_Registers_Kernel.h"
#define SALT 1130
#include "CUDA10_Registers_Kernel.h"
#define SALT 1131
#include "CUDA10_Registers_Kernel.h"
#define SALT 1132
#include "CUDA10_Registers_Kernel.h"
#define SALT 1133
#include "CUDA10_Registers_Kernel.h"
#define SALT 1134
#include "CUDA10_Registers_Kernel.h"
#define SALT 1135
#include "CUDA10_Registers_Kernel.h"
#define SALT 1136
#include "CUDA10_Registers_Kernel.h"
#define SALT 1137
#include "CUDA10_Registers_Kernel.h"
#define SALT 1138
#include "CUDA10_Registers_Kernel.h"
#define SALT 1139
#include "CUDA10_Registers_Kernel.h"
#define SALT 1140
#include "CUDA10_Registers_Kernel.h"
#define SALT 1141
#include "CUDA10_Registers_Kernel.h"
#define SALT 1142
#include "CUDA10_Registers_Kernel.h"
#define SALT 1143
#include "CUDA10_Registers_Kernel.h"
#define SALT 1144
#include "CUDA10_Registers_Kernel.h"
#define SALT 1145
#include "CUDA10_Registers_Kernel.h"
#define SALT 1146
#include "CUDA10_Registers_Kernel.h"
#define SALT 1147
#include "CUDA10_Registers_Kernel.h"
#define SALT 1148
#include "CUDA10_Registers_Kernel.h"
#define SALT 1149
#include "CUDA10_Registers_Kernel.h"
#define SALT 1150
#include "CUDA10_Registers_Kernel.h"
#define SALT 1151
#include "CUDA10_Registers_Kernel.h"
#define SALT 1152
#include "CUDA10_Registers_Kernel.h"
#define SALT 1153
#include "CUDA10_Registers_Kernel.h"
#define SALT 1154
#include "CUDA10_Registers_Kernel.h"
#define SALT 1155
#include "CUDA10_Registers_Kernel.h"
#define SALT 1156
#include "CUDA10_Registers_Kernel.h"
#define SALT 1157
#include "CUDA10_Registers_Kernel.h"
#define SALT 1158
#include "CUDA10_Registers_Kernel.h"
#define SALT 1159
#include "CUDA10_Registers_Kernel.h"
#define SALT 1160
#include "CUDA10_Registers_Kernel.h"
#define SALT 1161
#include "CUDA10_Registers_Kernel.h"
#define SALT 1162
#include "CUDA10_Registers_Kernel.h"
#define SALT 1163
#include "CUDA10_Registers_Kernel.h"
#define SALT 1164
#include "CUDA10_Registers_Kernel.h"
#define SALT 1165
#include "CUDA10_Registers_Kernel.h"
#define SALT 1166
#include "CUDA10_Registers_Kernel.h"
#define SALT 1167
#include "CUDA10_Registers_Kernel.h"
#define SALT 1168
#include "CUDA10_Registers_Kernel.h"
#define SALT 1169
#include "CUDA10_Registers_Kernel.h"
#define SALT 1170
#include "CUDA10_Registers_Kernel.h"
#define SALT 1171
#include "CUDA10_Registers_Kernel.h"
#define SALT 1172
#include "CUDA10_Registers_Kernel.h"
#define SALT 1173
#include "CUDA10_Registers_Kernel.h"
#define SALT 1174
#include "CUDA10_Registers_Kernel.h"
#define SALT 1175
#include "CUDA10_Registers_Kernel.h"
#define SALT 1176
#include "CUDA10_Registers_Kernel.h"
#define SALT 1177
#include "CUDA10_Registers_Kernel.h"
#define SALT 1178
#include "CUDA10_Registers_Kernel.h"
#define SALT 1179
#include "CUDA10_Registers_Kernel.h"
#define SALT 1180
#include "CUDA10_Registers_Kernel.h"
#define SALT 1181
#include "CUDA10_Registers_Kernel.h"
#define SALT 1182
#include "CUDA10_Registers_Kernel.h"
#define SALT 1183
#include "CUDA10_Registers_Kernel.h"
#define SALT 1184
#include "CUDA10_Registers_Kernel.h"
#define SALT 1185
#include "CUDA10_Registers_Kernel.h"
#define SALT 1186
#include "CUDA10_Registers_Kernel.h"
#define SALT 1187
#include "CUDA10_Registers_Kernel.h"
#define SALT 1188
#include "CUDA10_Registers_Kernel.h"
#define SALT 1189
#include "CUDA10_Registers_Kernel.h"
#define SALT 1190
#include "CUDA10_Registers_Kernel.h"
#define SALT 1191
#include "CUDA10_Registers_Kernel.h"
#define SALT 1192
#include "CUDA10_Registers_Kernel.h"
#define SALT 1193
#include "CUDA10_Registers_Kernel.h"
#define SALT 1194
#include "CUDA10_Registers_Kernel.h"
#define SALT 1195
#include "CUDA10_Registers_Kernel.h"
#define SALT 1196
#include "CUDA10_Registers_Kernel.h"
#define SALT 1197
#include "CUDA10_Registers_Kernel.h"
#define SALT 1198
#include "CUDA10_Registers_Kernel.h"
#define SALT 1199
#include "CUDA10_Registers_Kernel.h"
#define SALT 1200
#include "CUDA10_Registers_Kernel.h"
#define SALT 1201
#include "CUDA10_Registers_Kernel.h"
#define SALT 1202
#include "CUDA10_Registers_Kernel.h"
#define SALT 1203
#include "CUDA10_Registers_Kernel.h"
#define SALT 1204
#include "CUDA10_Registers_Kernel.h"
#define SALT 1205
#include "CUDA10_Registers_Kernel.h"
#define SALT 1206
#include "CUDA10_Registers_Kernel.h"
#define SALT 1207
#include "CUDA10_Registers_Kernel.h"
#define SALT 1208
#include "CUDA10_Registers_Kernel.h"
#define SALT 1209
#include "CUDA10_Registers_Kernel.h"
#define SALT 1210
#include "CUDA10_Registers_Kernel.h"
#define SALT 1211
#include "CUDA10_Registers_Kernel.h"
#define SALT 1212
#include "CUDA10_Registers_Kernel.h"
#define SALT 1213
#include "CUDA10_Registers_Kernel.h"
#define SALT 1214
#include "CUDA10_Registers_Kernel.h"
#define SALT 1215
#include "CUDA10_Registers_Kernel.h"
#define SALT 1216
#include "CUDA10_Registers_Kernel.h"
#define SALT 1217
#include "CUDA10_Registers_Kernel.h"
#define SALT 1218
#include "CUDA10_Registers_Kernel.h"
#define SALT 1219
#include "CUDA10_Registers_Kernel.h"
#define SALT 1220
#include "CUDA10_Registers_Kernel.h"
#define SALT 1221
#include "CUDA10_Registers_Kernel.h"
#define SALT 1222
#include "CUDA10_Registers_Kernel.h"
#define SALT 1223
#include "CUDA10_Registers_Kernel.h"
#define SALT 1224
#include "CUDA10_Registers_Kernel.h"
#define SALT 1225
#include "CUDA10_Registers_Kernel.h"
#define SALT 1226
#include "CUDA10_Registers_Kernel.h"
#define SALT 1227
#include "CUDA10_Registers_Kernel.h"
#define SALT 1228
#include "CUDA10_Registers_Kernel.h"
#define SALT 1229
#include "CUDA10_Registers_Kernel.h"
#define SALT 1230
#include "CUDA10_Registers_Kernel.h"
#define SALT 1231
#include "CUDA10_Registers_Kernel.h"
#define SALT 1232
#include "CUDA10_Registers_Kernel.h"
#define SALT 1233
#include "CUDA10_Registers_Kernel.h"
#define SALT 1234
#include "CUDA10_Registers_Kernel.h"
#define SALT 1235
#include "CUDA10_Registers_Kernel.h"
#define SALT 1236
#include "CUDA10_Registers_Kernel.h"
#define SALT 1237
#include "CUDA10_Registers_Kernel.h"
#define SALT 1238
#include "CUDA10_Registers_Kernel.h"
#define SALT 1239
#include "CUDA10_Registers_Kernel.h"
#define SALT 1240
#include "CUDA10_Registers_Kernel.h"
#define SALT 1241
#include "CUDA10_Registers_Kernel.h"
#define SALT 1242
#include "CUDA10_Registers_Kernel.h"
#define SALT 1243
#include "CUDA10_Registers_Kernel.h"
#define SALT 1244
#include "CUDA10_Registers_Kernel.h"
#define SALT 1245
#include "CUDA10_Registers_Kernel.h"
#define SALT 1246
#include "CUDA10_Registers_Kernel.h"
#define SALT 1247
#include "CUDA10_Registers_Kernel.h"
#define SALT 1248
#include "CUDA10_Registers_Kernel.h"
#define SALT 1249
#include "CUDA10_Registers_Kernel.h"
#define SALT 1250
#include "CUDA10_Registers_Kernel.h"
#define SALT 1251
#include "CUDA10_Registers_Kernel.h"
#define SALT 1252
#include "CUDA10_Registers_Kernel.h"
#define SALT 1253
#include "CUDA10_Registers_Kernel.h"
#define SALT 1254
#include "CUDA10_Registers_Kernel.h"
#define SALT 1255
#include "CUDA10_Registers_Kernel.h"
#define SALT 1256
#include "CUDA10_Registers_Kernel.h"
#define SALT 1257
#include "CUDA10_Registers_Kernel.h"
#define SALT 1258
#include "CUDA10_Registers_Kernel.h"
#define SALT 1259
#include "CUDA10_Registers_Kernel.h"
#define SALT 1260
#include "CUDA10_Registers_Kernel.h"
#define SALT 1261
#include "CUDA10_Registers_Kernel.h"
#define SALT 1262
#include "CUDA10_Registers_Kernel.h"
#define SALT 1263
#include "CUDA10_Registers_Kernel.h"
#define SALT 1264
#include "CUDA10_Registers_Kernel.h"
#define SALT 1265
#include "CUDA10_Registers_Kernel.h"
#define SALT 1266
#include "CUDA10_Registers_Kernel.h"
#define SALT 1267
#include "CUDA10_Registers_Kernel.h"
#define SALT 1268
#include "CUDA10_Registers_Kernel.h"
#define SALT 1269
#include "CUDA10_Registers_Kernel.h"
#define SALT 1270
#include "CUDA10_Registers_Kernel.h"
#define SALT 1271
#include "CUDA10_Registers_Kernel.h"
#define SALT 1272
#include "CUDA10_Registers_Kernel.h"
#define SALT 1273
#include "CUDA10_Registers_Kernel.h"
#define SALT 1274
#include "CUDA10_Registers_Kernel.h"
#define SALT 1275
#include "CUDA10_Registers_Kernel.h"
#define SALT 1276
#include "CUDA10_Registers_Kernel.h"
#define SALT 1277
#include "CUDA10_Registers_Kernel.h"
#define SALT 1278
#include "CUDA10_Registers_Kernel.h"
#define SALT 1279
#include "CUDA10_Registers_Kernel.h"
#define SALT 1280
#include "CUDA10_Registers_Kernel.h"
#define SALT 1281
#include "CUDA10_Registers_Kernel.h"
#define SALT 1282
#include "CUDA10_Registers_Kernel.h"
#define SALT 1283
#include "CUDA10_Registers_Kernel.h"
#define SALT 1284
#include "CUDA10_Registers_Kernel.h"
#define SALT 1285
#include "CUDA10_Registers_Kernel.h"
#define SALT 1286
#include "CUDA10_Registers_Kernel.h"
#define SALT 1287
#include "CUDA10_Registers_Kernel.h"
#define SALT 1288
#include "CUDA10_Registers_Kernel.h"
#define SALT 1289
#include "CUDA10_Registers_Kernel.h"
#define SALT 1290
#include "CUDA10_Registers_Kernel.h"
#define SALT 1291
#include "CUDA10_Registers_Kernel.h"
#define SALT 1292
#include "CUDA10_Registers_Kernel.h"
#define SALT 1293
#include "CUDA10_Registers_Kernel.h"
#define SALT 1294
#include "CUDA10_Registers_Kernel.h"
#define SALT 1295
#include "CUDA10_Registers_Kernel.h"
#define SALT 1296
#include "CUDA10_Registers_Kernel.h"
#define SALT 1297
#include "CUDA10_Registers_Kernel.h"
#define SALT 1298
#include "CUDA10_Registers_Kernel.h"
#define SALT 1299
#include "CUDA10_Registers_Kernel.h"
#define SALT 1300
#include "CUDA10_Registers_Kernel.h"
#define SALT 1301
#include "CUDA10_Registers_Kernel.h"
#define SALT 1302
#include "CUDA10_Registers_Kernel.h"
#define SALT 1303
#include "CUDA10_Registers_Kernel.h"
#define SALT 1304
#include "CUDA10_Registers_Kernel.h"
#define SALT 1305
#include "CUDA10_Registers_Kernel.h"
#define SALT 1306
#include "CUDA10_Registers_Kernel.h"
#define SALT 1307
#include "CUDA10_Registers_Kernel.h"
#define SALT 1308
#include "CUDA10_Registers_Kernel.h"
#define SALT 1309
#include "CUDA10_Registers_Kernel.h"
#define SALT 1310
#include "CUDA10_Registers_Kernel.h"
#define SALT 1311
#include "CUDA10_Registers_Kernel.h"
#define SALT 1312
#include "CUDA10_Registers_Kernel.h"
#define SALT 1313
#include "CUDA10_Registers_Kernel.h"
#define SALT 1314
#include "CUDA10_Registers_Kernel.h"
#define SALT 1315
#include "CUDA10_Registers_Kernel.h"
#define SALT 1316
#include "CUDA10_Registers_Kernel.h"
#define SALT 1317
#include "CUDA10_Registers_Kernel.h"
#define SALT 1318
#include "CUDA10_Registers_Kernel.h"
#define SALT 1319
#include "CUDA10_Registers_Kernel.h"
#define SALT 1320
#include "CUDA10_Registers_Kernel.h"
#define SALT 1321
#include "CUDA10_Registers_Kernel.h"
#define SALT 1322
#include "CUDA10_Registers_Kernel.h"
#define SALT 1323
#include "CUDA10_Registers_Kernel.h"
#define SALT 1324
#include "CUDA10_Registers_Kernel.h"
#define SALT 1325
#include "CUDA10_Registers_Kernel.h"
#define SALT 1326
#include "CUDA10_Registers_Kernel.h"
#define SALT 1327
#include "CUDA10_Registers_Kernel.h"
#define SALT 1328
#include "CUDA10_Registers_Kernel.h"
#define SALT 1329
#include "CUDA10_Registers_Kernel.h"
#define SALT 1330
#include "CUDA10_Registers_Kernel.h"
#define SALT 1331
#include "CUDA10_Registers_Kernel.h"
#define SALT 1332
#include "CUDA10_Registers_Kernel.h"
#define SALT 1333
#include "CUDA10_Registers_Kernel.h"
#define SALT 1334
#include "CUDA10_Registers_Kernel.h"
#define SALT 1335
#include "CUDA10_Registers_Kernel.h"
#define SALT 1336
#include "CUDA10_Registers_Kernel.h"
#define SALT 1337
#include "CUDA10_Registers_Kernel.h"
#define SALT 1338
#include "CUDA10_Registers_Kernel.h"
#define SALT 1339
#include "CUDA10_Registers_Kernel.h"
#define SALT 1340
#include "CUDA10_Registers_Kernel.h"
#define SALT 1341
#include "CUDA10_Registers_Kernel.h"
#define SALT 1342
#include "CUDA10_Registers_Kernel.h"
#define SALT 1343
#include "CUDA10_Registers_Kernel.h"
#define SALT 1344
#include "CUDA10_Registers_Kernel.h"
#define SALT 1345
#include "CUDA10_Registers_Kernel.h"
#define SALT 1346
#include "CUDA10_Registers_Kernel.h"
#define SALT 1347
#include "CUDA10_Registers_Kernel.h"
#define SALT 1348
#include "CUDA10_Registers_Kernel.h"
#define SALT 1349
#include "CUDA10_Registers_Kernel.h"
#define SALT 1350
#include "CUDA10_Registers_Kernel.h"
#define SALT 1351
#include "CUDA10_Registers_Kernel.h"
#define SALT 1352
#include "CUDA10_Registers_Kernel.h"
#define SALT 1353
#include "CUDA10_Registers_Kernel.h"
#define SALT 1354
#include "CUDA10_Registers_Kernel.h"
#define SALT 1355
#include "CUDA10_Registers_Kernel.h"
#define SALT 1356
#include "CUDA10_Registers_Kernel.h"
#define SALT 1357
#include "CUDA10_Registers_Kernel.h"
#define SALT 1358
#include "CUDA10_Registers_Kernel.h"
#define SALT 1359
#include "CUDA10_Registers_Kernel.h"
#define SALT 1360
#include "CUDA10_Registers_Kernel.h"
#define SALT 1361
#include "CUDA10_Registers_Kernel.h"
#define SALT 1362
#include "CUDA10_Registers_Kernel.h"
#define SALT 1363
#include "CUDA10_Registers_Kernel.h"
#define SALT 1364
#include "CUDA10_Registers_Kernel.h"
#define SALT 1365
#include "CUDA10_Registers_Kernel.h"
#define SALT 1366
#include "CUDA10_Registers_Kernel.h"
#define SALT 1367
#include "CUDA10_Registers_Kernel.h"
#define SALT 1368
#include "CUDA10_Registers_Kernel.h"
#define SALT 1369
#include "CUDA10_Registers_Kernel.h"
#define SALT 1370
#include "CUDA10_Registers_Kernel.h"
#define SALT 1371
#include "CUDA10_Registers_Kernel.h"
#define SALT 1372
#include "CUDA10_Registers_Kernel.h"
#define SALT 1373
#include "CUDA10_Registers_Kernel.h"
#define SALT 1374
#include "CUDA10_Registers_Kernel.h"
#define SALT 1375
#include "CUDA10_Registers_Kernel.h"
#define SALT 1376
#include "CUDA10_Registers_Kernel.h"
#define SALT 1377
#include "CUDA10_Registers_Kernel.h"
#define SALT 1378
#include "CUDA10_Registers_Kernel.h"
#define SALT 1379
#include "CUDA10_Registers_Kernel.h"
#define SALT 1380
#include "CUDA10_Registers_Kernel.h"
#define SALT 1381
#include "CUDA10_Registers_Kernel.h"
#define SALT 1382
#include "CUDA10_Registers_Kernel.h"
#define SALT 1383
#include "CUDA10_Registers_Kernel.h"
#define SALT 1384
#include "CUDA10_Registers_Kernel.h"
#define SALT 1385
#include "CUDA10_Registers_Kernel.h"
#define SALT 1386
#include "CUDA10_Registers_Kernel.h"
#define SALT 1387
#include "CUDA10_Registers_Kernel.h"
#define SALT 1388
#include "CUDA10_Registers_Kernel.h"
#define SALT 1389
#include "CUDA10_Registers_Kernel.h"
#define SALT 1390
#include "CUDA10_Registers_Kernel.h"
#define SALT 1391
#include "CUDA10_Registers_Kernel.h"
#define SALT 1392
#include "CUDA10_Registers_Kernel.h"
#define SALT 1393
#include "CUDA10_Registers_Kernel.h"
#define SALT 1394
#include "CUDA10_Registers_Kernel.h"
#define SALT 1395
#include "CUDA10_Registers_Kernel.h"
#define SALT 1396
#include "CUDA10_Registers_Kernel.h"
#define SALT 1397
#include "CUDA10_Registers_Kernel.h"
#define SALT 1398
#include "CUDA10_Registers_Kernel.h"
#define SALT 1399
#include "CUDA10_Registers_Kernel.h"
#define SALT 1400
#include "CUDA10_Registers_Kernel.h"
#define SALT 1401
#include "CUDA10_Registers_Kernel.h"
#define SALT 1402
#include "CUDA10_Registers_Kernel.h"
#define SALT 1403
#include "CUDA10_Registers_Kernel.h"
#define SALT 1404
#include "CUDA10_Registers_Kernel.h"
#define SALT 1405
#include "CUDA10_Registers_Kernel.h"
#define SALT 1406
#include "CUDA10_Registers_Kernel.h"
#define SALT 1407
#include "CUDA10_Registers_Kernel.h"
#define SALT 1408
#include "CUDA10_Registers_Kernel.h"
#define SALT 1409
#include "CUDA10_Registers_Kernel.h"
#define SALT 1410
#include "CUDA10_Registers_Kernel.h"
#define SALT 1411
#include "CUDA10_Registers_Kernel.h"
#define SALT 1412
#include "CUDA10_Registers_Kernel.h"
#define SALT 1413
#include "CUDA10_Registers_Kernel.h"
#define SALT 1414
#include "CUDA10_Registers_Kernel.h"
#define SALT 1415
#include "CUDA10_Registers_Kernel.h"
#define SALT 1416
#include "CUDA10_Registers_Kernel.h"
#define SALT 1417
#include "CUDA10_Registers_Kernel.h"
#define SALT 1418
#include "CUDA10_Registers_Kernel.h"
#define SALT 1419
#include "CUDA10_Registers_Kernel.h"
#define SALT 1420
#include "CUDA10_Registers_Kernel.h"
#define SALT 1421
#include "CUDA10_Registers_Kernel.h"
#define SALT 1422
#include "CUDA10_Registers_Kernel.h"
#define SALT 1423
#include "CUDA10_Registers_Kernel.h"
#define SALT 1424
#include "CUDA10_Registers_Kernel.h"
#define SALT 1425
#include "CUDA10_Registers_Kernel.h"
#define SALT 1426
#include "CUDA10_Registers_Kernel.h"
#define SALT 1427
#include "CUDA10_Registers_Kernel.h"
#define SALT 1428
#include "CUDA10_Registers_Kernel.h"
#define SALT 1429
#include "CUDA10_Registers_Kernel.h"
#define SALT 1430
#include "CUDA10_Registers_Kernel.h"
#define SALT 1431
#include "CUDA10_Registers_Kernel.h"
#define SALT 1432
#include "CUDA10_Registers_Kernel.h"
#define SALT 1433
#include "CUDA10_Registers_Kernel.h"
#define SALT 1434
#include "CUDA10_Registers_Kernel.h"
#define SALT 1435
#include "CUDA10_Registers_Kernel.h"
#define SALT 1436
#include "CUDA10_Registers_Kernel.h"
#define SALT 1437
#include "CUDA10_Registers_Kernel.h"
#define SALT 1438
#include "CUDA10_Registers_Kernel.h"
#define SALT 1439
#include "CUDA10_Registers_Kernel.h"
#define SALT 1440
#include "CUDA10_Registers_Kernel.h"
#define SALT 1441
#include "CUDA10_Registers_Kernel.h"
#define SALT 1442
#include "CUDA10_Registers_Kernel.h"
#define SALT 1443
#include "CUDA10_Registers_Kernel.h"
#define SALT 1444
#include "CUDA10_Registers_Kernel.h"
#define SALT 1445
#include "CUDA10_Registers_Kernel.h"
#define SALT 1446
#include "CUDA10_Registers_Kernel.h"
#define SALT 1447
#include "CUDA10_Registers_Kernel.h"
#define SALT 1448
#include "CUDA10_Registers_Kernel.h"
#define SALT 1449
#include "CUDA10_Registers_Kernel.h"
#define SALT 1450
#include "CUDA10_Registers_Kernel.h"
#define SALT 1451
#include "CUDA10_Registers_Kernel.h"
#define SALT 1452
#include "CUDA10_Registers_Kernel.h"
#define SALT 1453
#include "CUDA10_Registers_Kernel.h"
#define SALT 1454
#include "CUDA10_Registers_Kernel.h"
#define SALT 1455
#include "CUDA10_Registers_Kernel.h"
#define SALT 1456
#include "CUDA10_Registers_Kernel.h"
#define SALT 1457
#include "CUDA10_Registers_Kernel.h"
#define SALT 1458
#include "CUDA10_Registers_Kernel.h"
#define SALT 1459
#include "CUDA10_Registers_Kernel.h"
#define SALT 1460
#include "CUDA10_Registers_Kernel.h"
#define SALT 1461
#include "CUDA10_Registers_Kernel.h"
#define SALT 1462
#include "CUDA10_Registers_Kernel.h"
#define SALT 1463
#include "CUDA10_Registers_Kernel.h"
#define SALT 1464
#include "CUDA10_Registers_Kernel.h"
#define SALT 1465
#include "CUDA10_Registers_Kernel.h"
#define SALT 1466
#include "CUDA10_Registers_Kernel.h"
#define SALT 1467
#include "CUDA10_Registers_Kernel.h"
#define SALT 1468
#include "CUDA10_Registers_Kernel.h"
#define SALT 1469
#include "CUDA10_Registers_Kernel.h"
#define SALT 1470
#include "CUDA10_Registers_Kernel.h"
#define SALT 1471
#include "CUDA10_Registers_Kernel.h"
#define SALT 1472
#include "CUDA10_Registers_Kernel.h"
#define SALT 1473
#include "CUDA10_Registers_Kernel.h"
#define SALT 1474
#include "CUDA10_Registers_Kernel.h"
#define SALT 1475
#include "CUDA10_Registers_Kernel.h"
#define SALT 1476
#include "CUDA10_Registers_Kernel.h"
#define SALT 1477
#include "CUDA10_Registers_Kernel.h"
#define SALT 1478
#include "CUDA10_Registers_Kernel.h"
#define SALT 1479
#include "CUDA10_Registers_Kernel.h"
#define SALT 1480
#include "CUDA10_Registers_Kernel.h"
#define SALT 1481
#include "CUDA10_Registers_Kernel.h"
#define SALT 1482
#include "CUDA10_Registers_Kernel.h"
#define SALT 1483
#include "CUDA10_Registers_Kernel.h"
#define SALT 1484
#include "CUDA10_Registers_Kernel.h"
#define SALT 1485
#include "CUDA10_Registers_Kernel.h"
#define SALT 1486
#include "CUDA10_Registers_Kernel.h"
#define SALT 1487
#include "CUDA10_Registers_Kernel.h"
#define SALT 1488
#include "CUDA10_Registers_Kernel.h"
#define SALT 1489
#include "CUDA10_Registers_Kernel.h"
#define SALT 1490
#include "CUDA10_Registers_Kernel.h"
#define SALT 1491
#include "CUDA10_Registers_Kernel.h"
#define SALT 1492
#include "CUDA10_Registers_Kernel.h"
#define SALT 1493
#include "CUDA10_Registers_Kernel.h"
#define SALT 1494
#include "CUDA10_Registers_Kernel.h"
#define SALT 1495
#include "CUDA10_Registers_Kernel.h"
#define SALT 1496
#include "CUDA10_Registers_Kernel.h"
#define SALT 1497
#include "CUDA10_Registers_Kernel.h"
#define SALT 1498
#include "CUDA10_Registers_Kernel.h"
#define SALT 1499
#include "CUDA10_Registers_Kernel.h"
#define SALT 1500
#include "CUDA10_Registers_Kernel.h"
#define SALT 1501
#include "CUDA10_Registers_Kernel.h"
#define SALT 1502
#include "CUDA10_Registers_Kernel.h"
#define SALT 1503
#include "CUDA10_Registers_Kernel.h"
#define SALT 1504
#include "CUDA10_Registers_Kernel.h"
#define SALT 1505
#include "CUDA10_Registers_Kernel.h"
#define SALT 1506
#include "CUDA10_Registers_Kernel.h"
#define SALT 1507
#include "CUDA10_Registers_Kernel.h"
#define SALT 1508
#include "CUDA10_Registers_Kernel.h"
#define SALT 1509
#include "CUDA10_Registers_Kernel.h"
#define SALT 1510
#include "CUDA10_Registers_Kernel.h"
#define SALT 1511
#include "CUDA10_Registers_Kernel.h"
#define SALT 1512
#include "CUDA10_Registers_Kernel.h"
#define SALT 1513
#include "CUDA10_Registers_Kernel.h"
#define SALT 1514
#include "CUDA10_Registers_Kernel.h"
#define SALT 1515
#include "CUDA10_Registers_Kernel.h"
#define SALT 1516
#include "CUDA10_Registers_Kernel.h"
#define SALT 1517
#include "CUDA10_Registers_Kernel.h"
#define SALT 1518
#include "CUDA10_Registers_Kernel.h"
#define SALT 1519
#include "CUDA10_Registers_Kernel.h"
#define SALT 1520
#include "CUDA10_Registers_Kernel.h"
#define SALT 1521
#include "CUDA10_Registers_Kernel.h"
#define SALT 1522
#include "CUDA10_Registers_Kernel.h"
#define SALT 1523
#include "CUDA10_Registers_Kernel.h"
#define SALT 1524
#include "CUDA10_Registers_Kernel.h"
#define SALT 1525
#include "CUDA10_Registers_Kernel.h"
#define SALT 1526
#include "CUDA10_Registers_Kernel.h"
#define SALT 1527
#include "CUDA10_Registers_Kernel.h"
#define SALT 1528
#include "CUDA10_Registers_Kernel.h"
#define SALT 1529
#include "CUDA10_Registers_Kernel.h"
#define SALT 1530
#include "CUDA10_Registers_Kernel.h"
#define SALT 1531
#include "CUDA10_Registers_Kernel.h"
#define SALT 1532
#include "CUDA10_Registers_Kernel.h"
#define SALT 1533
#include "CUDA10_Registers_Kernel.h"
#define SALT 1534
#include "CUDA10_Registers_Kernel.h"
#define SALT 1535
#include "CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher2()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel2(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 1024: LAUNCH_KERNEL(1024); break;
	case 1025: LAUNCH_KERNEL(1025); break;
	case 1026: LAUNCH_KERNEL(1026); break;
	case 1027: LAUNCH_KERNEL(1027); break;
	case 1028: LAUNCH_KERNEL(1028); break;
	case 1029: LAUNCH_KERNEL(1029); break;
	case 1030: LAUNCH_KERNEL(1030); break;
	case 1031: LAUNCH_KERNEL(1031); break;
	case 1032: LAUNCH_KERNEL(1032); break;
	case 1033: LAUNCH_KERNEL(1033); break;
	case 1034: LAUNCH_KERNEL(1034); break;
	case 1035: LAUNCH_KERNEL(1035); break;
	case 1036: LAUNCH_KERNEL(1036); break;
	case 1037: LAUNCH_KERNEL(1037); break;
	case 1038: LAUNCH_KERNEL(1038); break;
	case 1039: LAUNCH_KERNEL(1039); break;
	case 1040: LAUNCH_KERNEL(1040); break;
	case 1041: LAUNCH_KERNEL(1041); break;
	case 1042: LAUNCH_KERNEL(1042); break;
	case 1043: LAUNCH_KERNEL(1043); break;
	case 1044: LAUNCH_KERNEL(1044); break;
	case 1045: LAUNCH_KERNEL(1045); break;
	case 1046: LAUNCH_KERNEL(1046); break;
	case 1047: LAUNCH_KERNEL(1047); break;
	case 1048: LAUNCH_KERNEL(1048); break;
	case 1049: LAUNCH_KERNEL(1049); break;
	case 1050: LAUNCH_KERNEL(1050); break;
	case 1051: LAUNCH_KERNEL(1051); break;
	case 1052: LAUNCH_KERNEL(1052); break;
	case 1053: LAUNCH_KERNEL(1053); break;
	case 1054: LAUNCH_KERNEL(1054); break;
	case 1055: LAUNCH_KERNEL(1055); break;
	case 1056: LAUNCH_KERNEL(1056); break;
	case 1057: LAUNCH_KERNEL(1057); break;
	case 1058: LAUNCH_KERNEL(1058); break;
	case 1059: LAUNCH_KERNEL(1059); break;
	case 1060: LAUNCH_KERNEL(1060); break;
	case 1061: LAUNCH_KERNEL(1061); break;
	case 1062: LAUNCH_KERNEL(1062); break;
	case 1063: LAUNCH_KERNEL(1063); break;
	case 1064: LAUNCH_KERNEL(1064); break;
	case 1065: LAUNCH_KERNEL(1065); break;
	case 1066: LAUNCH_KERNEL(1066); break;
	case 1067: LAUNCH_KERNEL(1067); break;
	case 1068: LAUNCH_KERNEL(1068); break;
	case 1069: LAUNCH_KERNEL(1069); break;
	case 1070: LAUNCH_KERNEL(1070); break;
	case 1071: LAUNCH_KERNEL(1071); break;
	case 1072: LAUNCH_KERNEL(1072); break;
	case 1073: LAUNCH_KERNEL(1073); break;
	case 1074: LAUNCH_KERNEL(1074); break;
	case 1075: LAUNCH_KERNEL(1075); break;
	case 1076: LAUNCH_KERNEL(1076); break;
	case 1077: LAUNCH_KERNEL(1077); break;
	case 1078: LAUNCH_KERNEL(1078); break;
	case 1079: LAUNCH_KERNEL(1079); break;
	case 1080: LAUNCH_KERNEL(1080); break;
	case 1081: LAUNCH_KERNEL(1081); break;
	case 1082: LAUNCH_KERNEL(1082); break;
	case 1083: LAUNCH_KERNEL(1083); break;
	case 1084: LAUNCH_KERNEL(1084); break;
	case 1085: LAUNCH_KERNEL(1085); break;
	case 1086: LAUNCH_KERNEL(1086); break;
	case 1087: LAUNCH_KERNEL(1087); break;
	case 1088: LAUNCH_KERNEL(1088); break;
	case 1089: LAUNCH_KERNEL(1089); break;
	case 1090: LAUNCH_KERNEL(1090); break;
	case 1091: LAUNCH_KERNEL(1091); break;
	case 1092: LAUNCH_KERNEL(1092); break;
	case 1093: LAUNCH_KERNEL(1093); break;
	case 1094: LAUNCH_KERNEL(1094); break;
	case 1095: LAUNCH_KERNEL(1095); break;
	case 1096: LAUNCH_KERNEL(1096); break;
	case 1097: LAUNCH_KERNEL(1097); break;
	case 1098: LAUNCH_KERNEL(1098); break;
	case 1099: LAUNCH_KERNEL(1099); break;
	case 1100: LAUNCH_KERNEL(1100); break;
	case 1101: LAUNCH_KERNEL(1101); break;
	case 1102: LAUNCH_KERNEL(1102); break;
	case 1103: LAUNCH_KERNEL(1103); break;
	case 1104: LAUNCH_KERNEL(1104); break;
	case 1105: LAUNCH_KERNEL(1105); break;
	case 1106: LAUNCH_KERNEL(1106); break;
	case 1107: LAUNCH_KERNEL(1107); break;
	case 1108: LAUNCH_KERNEL(1108); break;
	case 1109: LAUNCH_KERNEL(1109); break;
	case 1110: LAUNCH_KERNEL(1110); break;
	case 1111: LAUNCH_KERNEL(1111); break;
	case 1112: LAUNCH_KERNEL(1112); break;
	case 1113: LAUNCH_KERNEL(1113); break;
	case 1114: LAUNCH_KERNEL(1114); break;
	case 1115: LAUNCH_KERNEL(1115); break;
	case 1116: LAUNCH_KERNEL(1116); break;
	case 1117: LAUNCH_KERNEL(1117); break;
	case 1118: LAUNCH_KERNEL(1118); break;
	case 1119: LAUNCH_KERNEL(1119); break;
	case 1120: LAUNCH_KERNEL(1120); break;
	case 1121: LAUNCH_KERNEL(1121); break;
	case 1122: LAUNCH_KERNEL(1122); break;
	case 1123: LAUNCH_KERNEL(1123); break;
	case 1124: LAUNCH_KERNEL(1124); break;
	case 1125: LAUNCH_KERNEL(1125); break;
	case 1126: LAUNCH_KERNEL(1126); break;
	case 1127: LAUNCH_KERNEL(1127); break;
	case 1128: LAUNCH_KERNEL(1128); break;
	case 1129: LAUNCH_KERNEL(1129); break;
	case 1130: LAUNCH_KERNEL(1130); break;
	case 1131: LAUNCH_KERNEL(1131); break;
	case 1132: LAUNCH_KERNEL(1132); break;
	case 1133: LAUNCH_KERNEL(1133); break;
	case 1134: LAUNCH_KERNEL(1134); break;
	case 1135: LAUNCH_KERNEL(1135); break;
	case 1136: LAUNCH_KERNEL(1136); break;
	case 1137: LAUNCH_KERNEL(1137); break;
	case 1138: LAUNCH_KERNEL(1138); break;
	case 1139: LAUNCH_KERNEL(1139); break;
	case 1140: LAUNCH_KERNEL(1140); break;
	case 1141: LAUNCH_KERNEL(1141); break;
	case 1142: LAUNCH_KERNEL(1142); break;
	case 1143: LAUNCH_KERNEL(1143); break;
	case 1144: LAUNCH_KERNEL(1144); break;
	case 1145: LAUNCH_KERNEL(1145); break;
	case 1146: LAUNCH_KERNEL(1146); break;
	case 1147: LAUNCH_KERNEL(1147); break;
	case 1148: LAUNCH_KERNEL(1148); break;
	case 1149: LAUNCH_KERNEL(1149); break;
	case 1150: LAUNCH_KERNEL(1150); break;
	case 1151: LAUNCH_KERNEL(1151); break;
	case 1152: LAUNCH_KERNEL(1152); break;
	case 1153: LAUNCH_KERNEL(1153); break;
	case 1154: LAUNCH_KERNEL(1154); break;
	case 1155: LAUNCH_KERNEL(1155); break;
	case 1156: LAUNCH_KERNEL(1156); break;
	case 1157: LAUNCH_KERNEL(1157); break;
	case 1158: LAUNCH_KERNEL(1158); break;
	case 1159: LAUNCH_KERNEL(1159); break;
	case 1160: LAUNCH_KERNEL(1160); break;
	case 1161: LAUNCH_KERNEL(1161); break;
	case 1162: LAUNCH_KERNEL(1162); break;
	case 1163: LAUNCH_KERNEL(1163); break;
	case 1164: LAUNCH_KERNEL(1164); break;
	case 1165: LAUNCH_KERNEL(1165); break;
	case 1166: LAUNCH_KERNEL(1166); break;
	case 1167: LAUNCH_KERNEL(1167); break;
	case 1168: LAUNCH_KERNEL(1168); break;
	case 1169: LAUNCH_KERNEL(1169); break;
	case 1170: LAUNCH_KERNEL(1170); break;
	case 1171: LAUNCH_KERNEL(1171); break;
	case 1172: LAUNCH_KERNEL(1172); break;
	case 1173: LAUNCH_KERNEL(1173); break;
	case 1174: LAUNCH_KERNEL(1174); break;
	case 1175: LAUNCH_KERNEL(1175); break;
	case 1176: LAUNCH_KERNEL(1176); break;
	case 1177: LAUNCH_KERNEL(1177); break;
	case 1178: LAUNCH_KERNEL(1178); break;
	case 1179: LAUNCH_KERNEL(1179); break;
	case 1180: LAUNCH_KERNEL(1180); break;
	case 1181: LAUNCH_KERNEL(1181); break;
	case 1182: LAUNCH_KERNEL(1182); break;
	case 1183: LAUNCH_KERNEL(1183); break;
	case 1184: LAUNCH_KERNEL(1184); break;
	case 1185: LAUNCH_KERNEL(1185); break;
	case 1186: LAUNCH_KERNEL(1186); break;
	case 1187: LAUNCH_KERNEL(1187); break;
	case 1188: LAUNCH_KERNEL(1188); break;
	case 1189: LAUNCH_KERNEL(1189); break;
	case 1190: LAUNCH_KERNEL(1190); break;
	case 1191: LAUNCH_KERNEL(1191); break;
	case 1192: LAUNCH_KERNEL(1192); break;
	case 1193: LAUNCH_KERNEL(1193); break;
	case 1194: LAUNCH_KERNEL(1194); break;
	case 1195: LAUNCH_KERNEL(1195); break;
	case 1196: LAUNCH_KERNEL(1196); break;
	case 1197: LAUNCH_KERNEL(1197); break;
	case 1198: LAUNCH_KERNEL(1198); break;
	case 1199: LAUNCH_KERNEL(1199); break;
	case 1200: LAUNCH_KERNEL(1200); break;
	case 1201: LAUNCH_KERNEL(1201); break;
	case 1202: LAUNCH_KERNEL(1202); break;
	case 1203: LAUNCH_KERNEL(1203); break;
	case 1204: LAUNCH_KERNEL(1204); break;
	case 1205: LAUNCH_KERNEL(1205); break;
	case 1206: LAUNCH_KERNEL(1206); break;
	case 1207: LAUNCH_KERNEL(1207); break;
	case 1208: LAUNCH_KERNEL(1208); break;
	case 1209: LAUNCH_KERNEL(1209); break;
	case 1210: LAUNCH_KERNEL(1210); break;
	case 1211: LAUNCH_KERNEL(1211); break;
	case 1212: LAUNCH_KERNEL(1212); break;
	case 1213: LAUNCH_KERNEL(1213); break;
	case 1214: LAUNCH_KERNEL(1214); break;
	case 1215: LAUNCH_KERNEL(1215); break;
	case 1216: LAUNCH_KERNEL(1216); break;
	case 1217: LAUNCH_KERNEL(1217); break;
	case 1218: LAUNCH_KERNEL(1218); break;
	case 1219: LAUNCH_KERNEL(1219); break;
	case 1220: LAUNCH_KERNEL(1220); break;
	case 1221: LAUNCH_KERNEL(1221); break;
	case 1222: LAUNCH_KERNEL(1222); break;
	case 1223: LAUNCH_KERNEL(1223); break;
	case 1224: LAUNCH_KERNEL(1224); break;
	case 1225: LAUNCH_KERNEL(1225); break;
	case 1226: LAUNCH_KERNEL(1226); break;
	case 1227: LAUNCH_KERNEL(1227); break;
	case 1228: LAUNCH_KERNEL(1228); break;
	case 1229: LAUNCH_KERNEL(1229); break;
	case 1230: LAUNCH_KERNEL(1230); break;
	case 1231: LAUNCH_KERNEL(1231); break;
	case 1232: LAUNCH_KERNEL(1232); break;
	case 1233: LAUNCH_KERNEL(1233); break;
	case 1234: LAUNCH_KERNEL(1234); break;
	case 1235: LAUNCH_KERNEL(1235); break;
	case 1236: LAUNCH_KERNEL(1236); break;
	case 1237: LAUNCH_KERNEL(1237); break;
	case 1238: LAUNCH_KERNEL(1238); break;
	case 1239: LAUNCH_KERNEL(1239); break;
	case 1240: LAUNCH_KERNEL(1240); break;
	case 1241: LAUNCH_KERNEL(1241); break;
	case 1242: LAUNCH_KERNEL(1242); break;
	case 1243: LAUNCH_KERNEL(1243); break;
	case 1244: LAUNCH_KERNEL(1244); break;
	case 1245: LAUNCH_KERNEL(1245); break;
	case 1246: LAUNCH_KERNEL(1246); break;
	case 1247: LAUNCH_KERNEL(1247); break;
	case 1248: LAUNCH_KERNEL(1248); break;
	case 1249: LAUNCH_KERNEL(1249); break;
	case 1250: LAUNCH_KERNEL(1250); break;
	case 1251: LAUNCH_KERNEL(1251); break;
	case 1252: LAUNCH_KERNEL(1252); break;
	case 1253: LAUNCH_KERNEL(1253); break;
	case 1254: LAUNCH_KERNEL(1254); break;
	case 1255: LAUNCH_KERNEL(1255); break;
	case 1256: LAUNCH_KERNEL(1256); break;
	case 1257: LAUNCH_KERNEL(1257); break;
	case 1258: LAUNCH_KERNEL(1258); break;
	case 1259: LAUNCH_KERNEL(1259); break;
	case 1260: LAUNCH_KERNEL(1260); break;
	case 1261: LAUNCH_KERNEL(1261); break;
	case 1262: LAUNCH_KERNEL(1262); break;
	case 1263: LAUNCH_KERNEL(1263); break;
	case 1264: LAUNCH_KERNEL(1264); break;
	case 1265: LAUNCH_KERNEL(1265); break;
	case 1266: LAUNCH_KERNEL(1266); break;
	case 1267: LAUNCH_KERNEL(1267); break;
	case 1268: LAUNCH_KERNEL(1268); break;
	case 1269: LAUNCH_KERNEL(1269); break;
	case 1270: LAUNCH_KERNEL(1270); break;
	case 1271: LAUNCH_KERNEL(1271); break;
	case 1272: LAUNCH_KERNEL(1272); break;
	case 1273: LAUNCH_KERNEL(1273); break;
	case 1274: LAUNCH_KERNEL(1274); break;
	case 1275: LAUNCH_KERNEL(1275); break;
	case 1276: LAUNCH_KERNEL(1276); break;
	case 1277: LAUNCH_KERNEL(1277); break;
	case 1278: LAUNCH_KERNEL(1278); break;
	case 1279: LAUNCH_KERNEL(1279); break;
	case 1280: LAUNCH_KERNEL(1280); break;
	case 1281: LAUNCH_KERNEL(1281); break;
	case 1282: LAUNCH_KERNEL(1282); break;
	case 1283: LAUNCH_KERNEL(1283); break;
	case 1284: LAUNCH_KERNEL(1284); break;
	case 1285: LAUNCH_KERNEL(1285); break;
	case 1286: LAUNCH_KERNEL(1286); break;
	case 1287: LAUNCH_KERNEL(1287); break;
	case 1288: LAUNCH_KERNEL(1288); break;
	case 1289: LAUNCH_KERNEL(1289); break;
	case 1290: LAUNCH_KERNEL(1290); break;
	case 1291: LAUNCH_KERNEL(1291); break;
	case 1292: LAUNCH_KERNEL(1292); break;
	case 1293: LAUNCH_KERNEL(1293); break;
	case 1294: LAUNCH_KERNEL(1294); break;
	case 1295: LAUNCH_KERNEL(1295); break;
	case 1296: LAUNCH_KERNEL(1296); break;
	case 1297: LAUNCH_KERNEL(1297); break;
	case 1298: LAUNCH_KERNEL(1298); break;
	case 1299: LAUNCH_KERNEL(1299); break;
	case 1300: LAUNCH_KERNEL(1300); break;
	case 1301: LAUNCH_KERNEL(1301); break;
	case 1302: LAUNCH_KERNEL(1302); break;
	case 1303: LAUNCH_KERNEL(1303); break;
	case 1304: LAUNCH_KERNEL(1304); break;
	case 1305: LAUNCH_KERNEL(1305); break;
	case 1306: LAUNCH_KERNEL(1306); break;
	case 1307: LAUNCH_KERNEL(1307); break;
	case 1308: LAUNCH_KERNEL(1308); break;
	case 1309: LAUNCH_KERNEL(1309); break;
	case 1310: LAUNCH_KERNEL(1310); break;
	case 1311: LAUNCH_KERNEL(1311); break;
	case 1312: LAUNCH_KERNEL(1312); break;
	case 1313: LAUNCH_KERNEL(1313); break;
	case 1314: LAUNCH_KERNEL(1314); break;
	case 1315: LAUNCH_KERNEL(1315); break;
	case 1316: LAUNCH_KERNEL(1316); break;
	case 1317: LAUNCH_KERNEL(1317); break;
	case 1318: LAUNCH_KERNEL(1318); break;
	case 1319: LAUNCH_KERNEL(1319); break;
	case 1320: LAUNCH_KERNEL(1320); break;
	case 1321: LAUNCH_KERNEL(1321); break;
	case 1322: LAUNCH_KERNEL(1322); break;
	case 1323: LAUNCH_KERNEL(1323); break;
	case 1324: LAUNCH_KERNEL(1324); break;
	case 1325: LAUNCH_KERNEL(1325); break;
	case 1326: LAUNCH_KERNEL(1326); break;
	case 1327: LAUNCH_KERNEL(1327); break;
	case 1328: LAUNCH_KERNEL(1328); break;
	case 1329: LAUNCH_KERNEL(1329); break;
	case 1330: LAUNCH_KERNEL(1330); break;
	case 1331: LAUNCH_KERNEL(1331); break;
	case 1332: LAUNCH_KERNEL(1332); break;
	case 1333: LAUNCH_KERNEL(1333); break;
	case 1334: LAUNCH_KERNEL(1334); break;
	case 1335: LAUNCH_KERNEL(1335); break;
	case 1336: LAUNCH_KERNEL(1336); break;
	case 1337: LAUNCH_KERNEL(1337); break;
	case 1338: LAUNCH_KERNEL(1338); break;
	case 1339: LAUNCH_KERNEL(1339); break;
	case 1340: LAUNCH_KERNEL(1340); break;
	case 1341: LAUNCH_KERNEL(1341); break;
	case 1342: LAUNCH_KERNEL(1342); break;
	case 1343: LAUNCH_KERNEL(1343); break;
	case 1344: LAUNCH_KERNEL(1344); break;
	case 1345: LAUNCH_KERNEL(1345); break;
	case 1346: LAUNCH_KERNEL(1346); break;
	case 1347: LAUNCH_KERNEL(1347); break;
	case 1348: LAUNCH_KERNEL(1348); break;
	case 1349: LAUNCH_KERNEL(1349); break;
	case 1350: LAUNCH_KERNEL(1350); break;
	case 1351: LAUNCH_KERNEL(1351); break;
	case 1352: LAUNCH_KERNEL(1352); break;
	case 1353: LAUNCH_KERNEL(1353); break;
	case 1354: LAUNCH_KERNEL(1354); break;
	case 1355: LAUNCH_KERNEL(1355); break;
	case 1356: LAUNCH_KERNEL(1356); break;
	case 1357: LAUNCH_KERNEL(1357); break;
	case 1358: LAUNCH_KERNEL(1358); break;
	case 1359: LAUNCH_KERNEL(1359); break;
	case 1360: LAUNCH_KERNEL(1360); break;
	case 1361: LAUNCH_KERNEL(1361); break;
	case 1362: LAUNCH_KERNEL(1362); break;
	case 1363: LAUNCH_KERNEL(1363); break;
	case 1364: LAUNCH_KERNEL(1364); break;
	case 1365: LAUNCH_KERNEL(1365); break;
	case 1366: LAUNCH_KERNEL(1366); break;
	case 1367: LAUNCH_KERNEL(1367); break;
	case 1368: LAUNCH_KERNEL(1368); break;
	case 1369: LAUNCH_KERNEL(1369); break;
	case 1370: LAUNCH_KERNEL(1370); break;
	case 1371: LAUNCH_KERNEL(1371); break;
	case 1372: LAUNCH_KERNEL(1372); break;
	case 1373: LAUNCH_KERNEL(1373); break;
	case 1374: LAUNCH_KERNEL(1374); break;
	case 1375: LAUNCH_KERNEL(1375); break;
	case 1376: LAUNCH_KERNEL(1376); break;
	case 1377: LAUNCH_KERNEL(1377); break;
	case 1378: LAUNCH_KERNEL(1378); break;
	case 1379: LAUNCH_KERNEL(1379); break;
	case 1380: LAUNCH_KERNEL(1380); break;
	case 1381: LAUNCH_KERNEL(1381); break;
	case 1382: LAUNCH_KERNEL(1382); break;
	case 1383: LAUNCH_KERNEL(1383); break;
	case 1384: LAUNCH_KERNEL(1384); break;
	case 1385: LAUNCH_KERNEL(1385); break;
	case 1386: LAUNCH_KERNEL(1386); break;
	case 1387: LAUNCH_KERNEL(1387); break;
	case 1388: LAUNCH_KERNEL(1388); break;
	case 1389: LAUNCH_KERNEL(1389); break;
	case 1390: LAUNCH_KERNEL(1390); break;
	case 1391: LAUNCH_KERNEL(1391); break;
	case 1392: LAUNCH_KERNEL(1392); break;
	case 1393: LAUNCH_KERNEL(1393); break;
	case 1394: LAUNCH_KERNEL(1394); break;
	case 1395: LAUNCH_KERNEL(1395); break;
	case 1396: LAUNCH_KERNEL(1396); break;
	case 1397: LAUNCH_KERNEL(1397); break;
	case 1398: LAUNCH_KERNEL(1398); break;
	case 1399: LAUNCH_KERNEL(1399); break;
	case 1400: LAUNCH_KERNEL(1400); break;
	case 1401: LAUNCH_KERNEL(1401); break;
	case 1402: LAUNCH_KERNEL(1402); break;
	case 1403: LAUNCH_KERNEL(1403); break;
	case 1404: LAUNCH_KERNEL(1404); break;
	case 1405: LAUNCH_KERNEL(1405); break;
	case 1406: LAUNCH_KERNEL(1406); break;
	case 1407: LAUNCH_KERNEL(1407); break;
	case 1408: LAUNCH_KERNEL(1408); break;
	case 1409: LAUNCH_KERNEL(1409); break;
	case 1410: LAUNCH_KERNEL(1410); break;
	case 1411: LAUNCH_KERNEL(1411); break;
	case 1412: LAUNCH_KERNEL(1412); break;
	case 1413: LAUNCH_KERNEL(1413); break;
	case 1414: LAUNCH_KERNEL(1414); break;
	case 1415: LAUNCH_KERNEL(1415); break;
	case 1416: LAUNCH_KERNEL(1416); break;
	case 1417: LAUNCH_KERNEL(1417); break;
	case 1418: LAUNCH_KERNEL(1418); break;
	case 1419: LAUNCH_KERNEL(1419); break;
	case 1420: LAUNCH_KERNEL(1420); break;
	case 1421: LAUNCH_KERNEL(1421); break;
	case 1422: LAUNCH_KERNEL(1422); break;
	case 1423: LAUNCH_KERNEL(1423); break;
	case 1424: LAUNCH_KERNEL(1424); break;
	case 1425: LAUNCH_KERNEL(1425); break;
	case 1426: LAUNCH_KERNEL(1426); break;
	case 1427: LAUNCH_KERNEL(1427); break;
	case 1428: LAUNCH_KERNEL(1428); break;
	case 1429: LAUNCH_KERNEL(1429); break;
	case 1430: LAUNCH_KERNEL(1430); break;
	case 1431: LAUNCH_KERNEL(1431); break;
	case 1432: LAUNCH_KERNEL(1432); break;
	case 1433: LAUNCH_KERNEL(1433); break;
	case 1434: LAUNCH_KERNEL(1434); break;
	case 1435: LAUNCH_KERNEL(1435); break;
	case 1436: LAUNCH_KERNEL(1436); break;
	case 1437: LAUNCH_KERNEL(1437); break;
	case 1438: LAUNCH_KERNEL(1438); break;
	case 1439: LAUNCH_KERNEL(1439); break;
	case 1440: LAUNCH_KERNEL(1440); break;
	case 1441: LAUNCH_KERNEL(1441); break;
	case 1442: LAUNCH_KERNEL(1442); break;
	case 1443: LAUNCH_KERNEL(1443); break;
	case 1444: LAUNCH_KERNEL(1444); break;
	case 1445: LAUNCH_KERNEL(1445); break;
	case 1446: LAUNCH_KERNEL(1446); break;
	case 1447: LAUNCH_KERNEL(1447); break;
	case 1448: LAUNCH_KERNEL(1448); break;
	case 1449: LAUNCH_KERNEL(1449); break;
	case 1450: LAUNCH_KERNEL(1450); break;
	case 1451: LAUNCH_KERNEL(1451); break;
	case 1452: LAUNCH_KERNEL(1452); break;
	case 1453: LAUNCH_KERNEL(1453); break;
	case 1454: LAUNCH_KERNEL(1454); break;
	case 1455: LAUNCH_KERNEL(1455); break;
	case 1456: LAUNCH_KERNEL(1456); break;
	case 1457: LAUNCH_KERNEL(1457); break;
	case 1458: LAUNCH_KERNEL(1458); break;
	case 1459: LAUNCH_KERNEL(1459); break;
	case 1460: LAUNCH_KERNEL(1460); break;
	case 1461: LAUNCH_KERNEL(1461); break;
	case 1462: LAUNCH_KERNEL(1462); break;
	case 1463: LAUNCH_KERNEL(1463); break;
	case 1464: LAUNCH_KERNEL(1464); break;
	case 1465: LAUNCH_KERNEL(1465); break;
	case 1466: LAUNCH_KERNEL(1466); break;
	case 1467: LAUNCH_KERNEL(1467); break;
	case 1468: LAUNCH_KERNEL(1468); break;
	case 1469: LAUNCH_KERNEL(1469); break;
	case 1470: LAUNCH_KERNEL(1470); break;
	case 1471: LAUNCH_KERNEL(1471); break;
	case 1472: LAUNCH_KERNEL(1472); break;
	case 1473: LAUNCH_KERNEL(1473); break;
	case 1474: LAUNCH_KERNEL(1474); break;
	case 1475: LAUNCH_KERNEL(1475); break;
	case 1476: LAUNCH_KERNEL(1476); break;
	case 1477: LAUNCH_KERNEL(1477); break;
	case 1478: LAUNCH_KERNEL(1478); break;
	case 1479: LAUNCH_KERNEL(1479); break;
	case 1480: LAUNCH_KERNEL(1480); break;
	case 1481: LAUNCH_KERNEL(1481); break;
	case 1482: LAUNCH_KERNEL(1482); break;
	case 1483: LAUNCH_KERNEL(1483); break;
	case 1484: LAUNCH_KERNEL(1484); break;
	case 1485: LAUNCH_KERNEL(1485); break;
	case 1486: LAUNCH_KERNEL(1486); break;
	case 1487: LAUNCH_KERNEL(1487); break;
	case 1488: LAUNCH_KERNEL(1488); break;
	case 1489: LAUNCH_KERNEL(1489); break;
	case 1490: LAUNCH_KERNEL(1490); break;
	case 1491: LAUNCH_KERNEL(1491); break;
	case 1492: LAUNCH_KERNEL(1492); break;
	case 1493: LAUNCH_KERNEL(1493); break;
	case 1494: LAUNCH_KERNEL(1494); break;
	case 1495: LAUNCH_KERNEL(1495); break;
	case 1496: LAUNCH_KERNEL(1496); break;
	case 1497: LAUNCH_KERNEL(1497); break;
	case 1498: LAUNCH_KERNEL(1498); break;
	case 1499: LAUNCH_KERNEL(1499); break;
	case 1500: LAUNCH_KERNEL(1500); break;
	case 1501: LAUNCH_KERNEL(1501); break;
	case 1502: LAUNCH_KERNEL(1502); break;
	case 1503: LAUNCH_KERNEL(1503); break;
	case 1504: LAUNCH_KERNEL(1504); break;
	case 1505: LAUNCH_KERNEL(1505); break;
	case 1506: LAUNCH_KERNEL(1506); break;
	case 1507: LAUNCH_KERNEL(1507); break;
	case 1508: LAUNCH_KERNEL(1508); break;
	case 1509: LAUNCH_KERNEL(1509); break;
	case 1510: LAUNCH_KERNEL(1510); break;
	case 1511: LAUNCH_KERNEL(1511); break;
	case 1512: LAUNCH_KERNEL(1512); break;
	case 1513: LAUNCH_KERNEL(1513); break;
	case 1514: LAUNCH_KERNEL(1514); break;
	case 1515: LAUNCH_KERNEL(1515); break;
	case 1516: LAUNCH_KERNEL(1516); break;
	case 1517: LAUNCH_KERNEL(1517); break;
	case 1518: LAUNCH_KERNEL(1518); break;
	case 1519: LAUNCH_KERNEL(1519); break;
	case 1520: LAUNCH_KERNEL(1520); break;
	case 1521: LAUNCH_KERNEL(1521); break;
	case 1522: LAUNCH_KERNEL(1522); break;
	case 1523: LAUNCH_KERNEL(1523); break;
	case 1524: LAUNCH_KERNEL(1524); break;
	case 1525: LAUNCH_KERNEL(1525); break;
	case 1526: LAUNCH_KERNEL(1526); break;
	case 1527: LAUNCH_KERNEL(1527); break;
	case 1528: LAUNCH_KERNEL(1528); break;
	case 1529: LAUNCH_KERNEL(1529); break;
	case 1530: LAUNCH_KERNEL(1530); break;
	case 1531: LAUNCH_KERNEL(1531); break;
	case 1532: LAUNCH_KERNEL(1532); break;
	case 1533: LAUNCH_KERNEL(1533); break;
	case 1534: LAUNCH_KERNEL(1534); break;
	case 1535: LAUNCH_KERNEL(1535); break;
	default: ASSERT(FALSE);
	}
}

#endif
