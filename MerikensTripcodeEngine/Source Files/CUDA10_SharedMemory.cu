#include "hip/hip_runtime.h"
// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



// TO DO: Use smallChunkBitmap[]!



///////////////////////////////////////////////////////////////////////////////
// INCLUDE FILE(S)                                                           //
///////////////////////////////////////////////////////////////////////////////

#include "MerikensTripcodeEngine.h"



///////////////////////////////////////////////////////////////////////////////
// VARIABLES FOR CUDA CODES                                                  //
///////////////////////////////////////////////////////////////////////////////

__device__ __constant__ unsigned char cudaKeyCharTable_OneByte   [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char cudaKeyCharTable_FirstByte [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char cudaKeyCharTable_SecondByte[SIZE_KEY_CHAR_TABLE];
__device__ __constant__ char          CUDA_base64CharTable[64];
__device__ __constant__ unsigned char CUDA_smallChunkBitmap[SMALL_CHUNK_BITMAP_SIZE];



///////////////////////////////////////////////////////////////////////////////
// BITSLICE DES                                                              //
///////////////////////////////////////////////////////////////////////////////

#define NUM_THREADS_PER_BITSICE_DES   4

// FOR DEVICE CODES ONLY
#if   __CUDA_ARCH__ == 200
#define CUDA_DES_NUM_THREADS_PER_BLOCK      768
#elif __CUDA_ARCH__ == 300
#define CUDA_DES_NUM_THREADS_PER_BLOCK      768
#elif __CUDA_ARCH__ == 320
#define CUDA_DES_NUM_THREADS_PER_BLOCK      768
#elif __CUDA_ARCH__ == 350
#define CUDA_DES_NUM_THREADS_PER_BLOCK      768
#elif __CUDA_ARCH__ == 370
#define CUDA_DES_NUM_THREADS_PER_BLOCK      448
#elif __CUDA_ARCH__ == 500
#define CUDA_DES_NUM_THREADS_PER_BLOCK      512
#elif __CUDA_ARCH__ == 520
#define CUDA_DES_NUM_THREADS_PER_BLOCK      512
#elif __CUDA_ARCH__ == 530
#define CUDA_DES_NUM_THREADS_PER_BLOCK      512
#else
#define CUDA_DES_NUM_THREADS_PER_BLOCK      512 // dummy value to make nvcc happy
#endif
#define CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK (CUDA_DES_NUM_THREADS_PER_BLOCK / NUM_THREADS_PER_BITSICE_DES)
#define N CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK

#define CUDA_DES_BS_DEPTH                   32
#define CUDA_DES_MAX_PASS_COUNT             16

typedef int           DES_ARCH_WORD;
typedef int           DES_ARCH_WORD_32;
#define DES_ARCH_SIZE 4
#define DES_ARCH_BITS 32

typedef int           DES_Vector;
// #define CUDA_DES_BS_DEPTH  DES_ARCH_BITS
#define DES_VECTOR_ZERO               0
#define DES_VECTOR_ONES               ~(DES_Vector)0

#define DES_VECTOR_NOT(dst, a)        (dst) =  ~(a)
#define DES_VECTOR_AND(dst, a, b)     (dst) =   (a) &  (b)
#define DES_VECTOR_OR(dst, a, b)      (dst) =   (a) |  (b)
#define DES_VECTOR_AND_NOT(dst, a, b) (dst) =   (a) & ~(b)
#define DES_VECTOR_XOR_NOT(dst, a, b) (dst) = ~((a) ^  (b))
#define DES_VECTOR_NOT_OR(dst, a, b)  (dst) = ~((a) |  (b))
#define DES_VECTOR_SEL(dst, a, b, c)  (dst) = (((a) & ~(c)) ^ ((b) & (c)))
#define DES_VECTOR_XOR_FUNC(a, b)              ((a) ^  (b))
#define DES_VECTOR_XOR(dst, a, b)     (dst) = DES_VECTOR_XOR_FUNC((a), (b))
#define DES_VECTOR_SET(dst, ofs, src) *((DES_Vector *)((DES_Vector *)&(dst) + ((ofs) * N))) = (src)

#define DES_CONSTANT_QUALIFIERS      __device__ __constant__
#define DES_FUNCTION_QUALIFIERS      __device__ __forceinline__
#define DES_SBOX_FUNCTION_QUALIFIERS __device__ __forceinline__

extern __shared__ DES_Vector dataBlocks[];

const unsigned char expansionTable[48] = {
	31,  0,  1,  2,  3,  4,
	 3,  4,  5,  6,  7,  8,
	 7,  8,  9, 10, 11, 12,
	11, 12, 13, 14, 15, 16,
	15, 16, 17, 18, 19, 20,
	19, 20, 21, 22, 23, 24,
	23, 24, 25, 26, 27, 28,
	27, 28, 29, 30, 31,  0
};

const char charToIndexTableForDES[0x100] = {
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x00, 0x01,
	0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09,
	0x0a, 0x0b, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x0c, 0x0d, 0x0e, 0x0f, 0x10, 0x11, 0x12,
	0x13, 0x14, 0x15, 0x16, 0x17, 0x18, 0x19, 0x1a,
	0x1b, 0x1c, 0x1d, 0x1e, 0x1f, 0x20, 0x21, 0x22,
	0x23, 0x24, 0x25, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x26, 0x27, 0x28, 0x29, 0x2a, 0x2b, 0x2c,
	0x2d, 0x2e, 0x2f, 0x30, 0x31, 0x32, 0x33, 0x34,
	0x35, 0x36, 0x37, 0x38, 0x39, 0x3a, 0x3b, 0x3c,
	0x3d, 0x3e, 0x3f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
};

DES_CONSTANT_QUALIFIERS char CUDA_DES_indexToCharTable[64] =
//	"./0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
{
	/* 00 */ '.', '/',
	/* 02 */ '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 
	/* 12 */ 'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 
	/* 28 */ 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
	/* 38 */ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p',
	/* 54 */ 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z', 
};

DES_CONSTANT_QUALIFIERS unsigned char keySchedule[DES_SIZE_KEY_SCHEDULE] = {
	12, 46, 33, 52, 48, 20, 34, 55,  5, 13, 18, 40,  4, 32, 26, 27,
	38, 54, 53,  6, 31, 25, 19, 41, 15, 24, 28, 43, 30,  3, 35, 22,
	 2, 44, 14, 23, 51, 16, 29, 49,  7, 17, 37,  8,  9, 50, 42, 21,
	 5, 39, 26, 45, 41, 13, 27, 48, 53,  6, 11, 33, 52, 25, 19, 20,
	31, 47, 46, 54, 55, 18, 12, 34,  8, 17, 21, 36, 23, 49, 28, 15,
	24, 37,  7, 16, 44,  9, 22, 42,  0, 10, 30,  1,  2, 43, 35, 14,
	46, 25, 12, 31, 27, 54, 13, 34, 39, 47, 52, 19, 38, 11,  5,  6,
	48, 33, 32, 40, 41,  4, 53, 20, 51,  3,  7, 22,  9, 35, 14,  1,
	10, 23, 50,  2, 30, 24,  8, 28, 43, 49, 16, 44, 17, 29, 21,  0,
	32, 11, 53, 48, 13, 40, 54, 20, 25, 33, 38,  5, 55, 52, 46, 47,
	34, 19, 18, 26, 27, 45, 39,  6, 37, 42, 50,  8, 24, 21,  0, 44,
	49,  9, 36, 17, 16, 10, 51, 14, 29, 35,  2, 30,  3, 15,  7, 43,
	18, 52, 39, 34, 54, 26, 40,  6, 11, 19, 55, 46, 41, 38, 32, 33,
	20,  5,  4, 12, 13, 31, 25, 47, 23, 28, 36, 51, 10,  7, 43, 30,
	35, 24, 22,  3,  2, 49, 37,  0, 15, 21, 17, 16, 42,  1, 50, 29,
	 4, 38, 25, 20, 40, 12, 26, 47, 52,  5, 41, 32, 27, 55, 18, 19,
	 6, 46, 45, 53, 54, 48, 11, 33,  9, 14, 22, 37, 49, 50, 29, 16,
	21, 10,  8, 42, 17, 35, 23, 43,  1,  7,  3,  2, 28, 44, 36, 15,
	45, 55, 11,  6, 26, 53, 12, 33, 38, 46, 27, 18, 13, 41,  4,  5,
	47, 32, 31, 39, 40, 34, 52, 19, 24,  0,  8, 23, 35, 36, 15,  2,
	 7, 49, 51, 28,  3, 21,  9, 29, 44, 50, 42, 17, 14, 30, 22,  1,
	31, 41, 52, 47, 12, 39, 53, 19, 55, 32, 13,  4, 54, 27, 45, 46,
	33, 18, 48, 25, 26, 20, 38,  5, 10, 43, 51,  9, 21, 22,  1, 17,
	50, 35, 37, 14, 42,  7, 24, 15, 30, 36, 28,  3,  0, 16,  8, 44,
	55, 34, 45, 40,  5, 32, 46, 12, 48, 25,  6, 52, 47, 20, 38, 39,
	26, 11, 41, 18, 19, 13, 31, 53,  3, 36, 44,  2, 14, 15, 51, 10,
	43, 28, 30,  7, 35,  0, 17,  8, 23, 29, 21, 49, 50,  9,  1, 37,
	41, 20, 31, 26, 46, 18, 32, 53, 34, 11, 47, 38, 33,  6, 55, 25,
	12, 52, 27,  4,  5, 54, 48, 39, 42, 22, 30, 17,  0,  1, 37, 49,
	29, 14, 16, 50, 21, 43,  3, 51,  9, 15,  7, 35, 36, 24, 44, 23,
	27,  6, 48, 12, 32,  4, 18, 39, 20, 52, 33, 55, 19, 47, 41, 11,
	53, 38, 13, 45, 46, 40, 34, 25, 28,  8, 16,  3, 43, 44, 23, 35,
	15,  0,  2, 36,  7, 29, 42, 37, 24,  1, 50, 21, 22, 10, 30,  9,
	13, 47, 34, 53, 18, 45,  4, 25,  6, 38, 19, 41,  5, 33, 27, 52,
	39, 55, 54, 31, 32, 26, 20, 11, 14, 51,  2, 42, 29, 30,  9, 21,
	 1, 43, 17, 22, 50, 15, 28, 23, 10, 44, 36,  7,  8, 49, 16, 24,
	54, 33, 20, 39,  4, 31, 45, 11, 47, 55,  5, 27, 46, 19, 13, 38,
	25, 41, 40, 48, 18, 12,  6, 52,  0, 37, 17, 28, 15, 16, 24,  7,
	44, 29,  3,  8, 36,  1, 14,  9, 49, 30, 22, 50, 51, 35,  2, 10,
	40, 19,  6, 25, 45, 48, 31, 52, 33, 41, 46, 13, 32,  5, 54, 55,
	11, 27, 26, 34,  4, 53, 47, 38, 43, 23,  3, 14,  1,  2, 10, 50,
	30, 15, 42, 51, 22, 44,  0, 24, 35, 16,  8, 36, 37, 21, 17, 49,
	26,  5, 47, 11, 31, 34, 48, 38, 19, 27, 32, 54, 18, 46, 40, 41,
	52, 13, 12, 20, 45, 39, 33, 55, 29,  9, 42,  0, 44, 17, 49, 36,
	16,  1, 28, 37,  8, 30, 43, 10, 21,  2, 51, 22, 23,  7,  3, 35,
	19, 53, 40,  4, 55, 27, 41, 31, 12, 20, 25, 47, 11, 39, 33, 34,
	45,  6,  5, 13, 38, 32, 26, 48, 22,  2, 35, 50, 37, 10, 42, 29,
	 9, 51, 21, 30,  1, 23, 36,  3, 14, 24, 44, 15, 16,  0, 49, 28,
};

void DES_CreateExpansionFunction(char *saltString, unsigned char *expansionFunction)
{
	unsigned char saltChar1 = '.', saltChar2 = '.';
	DES_ARCH_WORD salt;
	DES_ARCH_WORD mask;
	int src, dst;

	if (saltString[0]) {
		saltChar1 = saltString[0];
		if (saltString[1])
			saltChar2 = saltString[1];
	}
	salt =    charToIndexTableForDES[saltChar1]
	       | (charToIndexTableForDES[saltChar2] << 6);

	mask = 1;
	for (dst = 0; dst < 48; dst++) {
		if (dst == 24) mask = 1;

		if (salt & mask) {
			if (dst < 24) src = dst + 24; else src = dst - 24;
		} else src = dst;

		expansionFunction[dst     ] = expansionTable[src];
		expansionFunction[dst + 48] = expansionTable[src] + 32;

		mask <<= 1;
	}
}

#include "CUDA10_S-boxes.h"

#define CLEAR_BLOCK_8(i)                                                             \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 0, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 1, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 2, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 3, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 4, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 5, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 6, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(dataBlocks[threadIdx.x + (i*N)] , 7, DES_VECTOR_ZERO); \

#define CLEAR_BLOCK()  \
	CLEAR_BLOCK_8(0);  \
	CLEAR_BLOCK_8(8);  \
	CLEAR_BLOCK_8(16); \
	CLEAR_BLOCK_8(24); \
	CLEAR_BLOCK_8(32); \
	CLEAR_BLOCK_8(40); \
	CLEAR_BLOCK_8(48); \
	CLEAR_BLOCK_8(56); \

DES_FUNCTION_QUALIFIERS
void DES_Crypt(volatile unsigned int keyFrom00To27, volatile unsigned int keyFrom28To48, unsigned char *CUDA_expansionFunction, DES_Vector *CUDA_keyFrom49To55Array)
{
	if (threadIdx.y == 0)
		CLEAR_BLOCK();
	
	DES_Vector *db = dataBlocks + threadIdx.x;
	int E0, E1, E2, E3, E4, E5;

	switch (threadIdx.y) {
	case 0: 
		E0 = CUDA_expansionFunction[0]*N;
		E1 = CUDA_expansionFunction[1]*N;
		E2 = CUDA_expansionFunction[2]*N;
		E3 = CUDA_expansionFunction[3]*N;
		E4 = CUDA_expansionFunction[4]*N;
		E5 = CUDA_expansionFunction[5]*N;
		break;
	case 1: 
		E0 = CUDA_expansionFunction[6]*N;
		E1 = CUDA_expansionFunction[7]*N;
		E2 = CUDA_expansionFunction[8]*N;
		E3 = CUDA_expansionFunction[9]*N;
		E4 = CUDA_expansionFunction[10]*N;
		E5 = CUDA_expansionFunction[11]*N;
		break;
	case 2: 
		E0 = CUDA_expansionFunction[24]*N;
		E1 = CUDA_expansionFunction[25]*N;
		E2 = CUDA_expansionFunction[26]*N;
		E3 = CUDA_expansionFunction[27]*N;
		E4 = CUDA_expansionFunction[28]*N;
		E5 = CUDA_expansionFunction[29]*N;
		break;
	case 3: 
		E0 = CUDA_expansionFunction[30]*N;
		E1 = CUDA_expansionFunction[31]*N;
		E2 = CUDA_expansionFunction[32]*N;
		E3 = CUDA_expansionFunction[33]*N;
		E4 = CUDA_expansionFunction[34]*N;
		E5 = CUDA_expansionFunction[35]*N;
		break;
	}
	
#define K00 ((keyFrom00To27 & (0x1U << 0)) ? 0xffffffffU : 0x0)
#define K01 ((keyFrom00To27 & (0x1U << 1)) ? 0xffffffffU : 0x0)
#define K02 ((keyFrom00To27 & (0x1U << 2)) ? 0xffffffffU : 0x0)
#define K03 ((keyFrom00To27 & (0x1U << 3)) ? 0xffffffffU : 0x0)
#define K04 ((keyFrom00To27 & (0x1U << 4)) ? 0xffffffffU : 0x0)
#define K05 ((keyFrom00To27 & (0x1U << 5)) ? 0xffffffffU : 0x0)
#define K06 ((keyFrom00To27 & (0x1U << 6)) ? 0xffffffffU : 0x0)
#define K07 ((keyFrom00To27 & (0x1U << 7)) ? 0xffffffffU : 0x0)
#define K08 ((keyFrom00To27 & (0x1U << 8)) ? 0xffffffffU : 0x0)
#define K09 ((keyFrom00To27 & (0x1U << 9)) ? 0xffffffffU : 0x0)
#define K10 ((keyFrom00To27 & (0x1U << 10)) ? 0xffffffffU : 0x0)
#define K11 ((keyFrom00To27 & (0x1U << 11)) ? 0xffffffffU : 0x0)
#define K12 ((keyFrom00To27 & (0x1U << 12)) ? 0xffffffffU : 0x0)
#define K13 ((keyFrom00To27 & (0x1U << 13)) ? 0xffffffffU : 0x0)
#define K14 ((keyFrom00To27 & (0x1U << 14)) ? 0xffffffffU : 0x0)
#define K15 ((keyFrom00To27 & (0x1U << 15)) ? 0xffffffffU : 0x0)
#define K16 ((keyFrom00To27 & (0x1U << 16)) ? 0xffffffffU : 0x0)
#define K17 ((keyFrom00To27 & (0x1U << 17)) ? 0xffffffffU : 0x0)
#define K18 ((keyFrom00To27 & (0x1U << 18)) ? 0xffffffffU : 0x0)
#define K19 ((keyFrom00To27 & (0x1U << 19)) ? 0xffffffffU : 0x0)
#define K20 ((keyFrom00To27 & (0x1U << 20)) ? 0xffffffffU : 0x0)
#define K21 ((keyFrom00To27 & (0x1U << 21)) ? 0xffffffffU : 0x0)
#define K22 ((keyFrom00To27 & (0x1U << 22)) ? 0xffffffffU : 0x0)
#define K23 ((keyFrom00To27 & (0x1U << 23)) ? 0xffffffffU : 0x0)
#define K24 ((keyFrom00To27 & (0x1U << 24)) ? 0xffffffffU : 0x0)
#define K25 ((keyFrom00To27 & (0x1U << 25)) ? 0xffffffffU : 0x0)
#define K26 ((keyFrom00To27 & (0x1U << 26)) ? 0xffffffffU : 0x0)
#define K27 ((keyFrom00To27 & (0x1U << 27)) ? 0xffffffffU : 0x0)
#define K28 ((keyFrom28To48 & (0x1U << (28 - 28))) ? 0xffffffffU : 0x0)
#define K29 ((keyFrom28To48 & (0x1U << (29 - 28))) ? 0xffffffffU : 0x0)
#define K30 ((keyFrom28To48 & (0x1U << (30 - 28))) ? 0xffffffffU : 0x0)
#define K31 ((keyFrom28To48 & (0x1U << (31 - 28))) ? 0xffffffffU : 0x0)
#define K32 ((keyFrom28To48 & (0x1U << (32 - 28))) ? 0xffffffffU : 0x0)
#define K33 ((keyFrom28To48 & (0x1U << (33 - 28))) ? 0xffffffffU : 0x0)
#define K34 ((keyFrom28To48 & (0x1U << (34 - 28))) ? 0xffffffffU : 0x0)
#define K35 ((keyFrom28To48 & (0x1U << (35 - 28))) ? 0xffffffffU : 0x0)
#define K36 ((keyFrom28To48 & (0x1U << (36 - 28))) ? 0xffffffffU : 0x0)
#define K37 ((keyFrom28To48 & (0x1U << (37 - 28))) ? 0xffffffffU : 0x0)
#define K38 ((keyFrom28To48 & (0x1U << (38 - 28))) ? 0xffffffffU : 0x0)
#define K39 ((keyFrom28To48 & (0x1U << (39 - 28))) ? 0xffffffffU : 0x0)
#define K40 ((keyFrom28To48 & (0x1U << (40 - 28))) ? 0xffffffffU : 0x0)
#define K41 ((keyFrom28To48 & (0x1U << (41 - 28))) ? 0xffffffffU : 0x0)
#define K42 ((keyFrom28To48 & (0x1U << (42 - 28))) ? 0xffffffffU : 0x0)
#define K43 ((keyFrom28To48 & (0x1U << (43 - 28))) ? 0xffffffffU : 0x0)
#define K44 ((keyFrom28To48 & (0x1U << (44 - 28))) ? 0xffffffffU : 0x0)
#define K45 ((keyFrom28To48 & (0x1U << (45 - 28))) ? 0xffffffffU : 0x0)
#define K46 ((keyFrom28To48 & (0x1U << (46 - 28))) ? 0xffffffffU : 0x0)
#define K47 ((keyFrom28To48 & (0x1U << (47 - 28))) ? 0xffffffffU : 0x0)
#define K48 ((keyFrom28To48 & (0x1U << (48 - 28))) ? 0xffffffffU : 0x0)

#define K00XOR(val) ((keyFrom00To27 & (0x1U << 0)) ? ~(val) : (val))
#define K01XOR(val) ((keyFrom00To27 & (0x1U << 1)) ? ~(val) : (val))
#define K02XOR(val) ((keyFrom00To27 & (0x1U << 2)) ? ~(val) : (val))
#define K03XOR(val) ((keyFrom00To27 & (0x1U << 3)) ? ~(val) : (val))
#define K04XOR(val) ((keyFrom00To27 & (0x1U << 4)) ? ~(val) : (val))
#define K05XOR(val) ((keyFrom00To27 & (0x1U << 5)) ? ~(val) : (val))
#define K06XOR(val) ((keyFrom00To27 & (0x1U << 6)) ? ~(val) : (val))
#define K07XOR(val) ((keyFrom00To27 & (0x1U << 7)) ? ~(val) : (val))
#define K08XOR(val) ((keyFrom00To27 & (0x1U << 8)) ? ~(val) : (val))
#define K09XOR(val) ((keyFrom00To27 & (0x1U << 9)) ? ~(val) : (val))
#define K10XOR(val) ((keyFrom00To27 & (0x1U << 10)) ? ~(val) : (val))
#define K11XOR(val) ((keyFrom00To27 & (0x1U << 11)) ? ~(val) : (val))
#define K12XOR(val) ((keyFrom00To27 & (0x1U << 12)) ? ~(val) : (val))
#define K13XOR(val) ((keyFrom00To27 & (0x1U << 13)) ? ~(val) : (val))
#define K14XOR(val) ((keyFrom00To27 & (0x1U << 14)) ? ~(val) : (val))
#define K15XOR(val) ((keyFrom00To27 & (0x1U << 15)) ? ~(val) : (val))
#define K16XOR(val) ((keyFrom00To27 & (0x1U << 16)) ? ~(val) : (val))
#define K17XOR(val) ((keyFrom00To27 & (0x1U << 17)) ? ~(val) : (val))
#define K18XOR(val) ((keyFrom00To27 & (0x1U << 18)) ? ~(val) : (val))
#define K19XOR(val) ((keyFrom00To27 & (0x1U << 19)) ? ~(val) : (val))
#define K20XOR(val) ((keyFrom00To27 & (0x1U << 20)) ? ~(val) : (val))
#define K21XOR(val) ((keyFrom00To27 & (0x1U << 21)) ? ~(val) : (val))
#define K22XOR(val) ((keyFrom00To27 & (0x1U << 22)) ? ~(val) : (val))
#define K23XOR(val) ((keyFrom00To27 & (0x1U << 23)) ? ~(val) : (val))
#define K24XOR(val) ((keyFrom00To27 & (0x1U << 24)) ? ~(val) : (val))
#define K25XOR(val) ((keyFrom00To27 & (0x1U << 25)) ? ~(val) : (val))
#define K26XOR(val) ((keyFrom00To27 & (0x1U << 26)) ? ~(val) : (val))
#define K27XOR(val) ((keyFrom00To27 & (0x1U << 27)) ? ~(val) : (val))
#define K28XOR(val) ((keyFrom28To48 & (0x1U << (28 - 28))) ? ~(val) : (val))
#define K29XOR(val) ((keyFrom28To48 & (0x1U << (29 - 28))) ? ~(val) : (val))
#define K30XOR(val) ((keyFrom28To48 & (0x1U << (30 - 28))) ? ~(val) : (val))
#define K31XOR(val) ((keyFrom28To48 & (0x1U << (31 - 28))) ? ~(val) : (val))
#define K32XOR(val) ((keyFrom28To48 & (0x1U << (32 - 28))) ? ~(val) : (val))
#define K33XOR(val) ((keyFrom28To48 & (0x1U << (33 - 28))) ? ~(val) : (val))
#define K34XOR(val) ((keyFrom28To48 & (0x1U << (34 - 28))) ? ~(val) : (val))
#define K35XOR(val) ((keyFrom28To48 & (0x1U << (35 - 28))) ? ~(val) : (val))
#define K36XOR(val) ((keyFrom28To48 & (0x1U << (36 - 28))) ? ~(val) : (val))
#define K37XOR(val) ((keyFrom28To48 & (0x1U << (37 - 28))) ? ~(val) : (val))
#define K38XOR(val) ((keyFrom28To48 & (0x1U << (38 - 28))) ? ~(val) : (val))
#define K39XOR(val) ((keyFrom28To48 & (0x1U << (39 - 28))) ? ~(val) : (val))
#define K40XOR(val) ((keyFrom28To48 & (0x1U << (40 - 28))) ? ~(val) : (val))
#define K41XOR(val) ((keyFrom28To48 & (0x1U << (41 - 28))) ? ~(val) : (val))
#define K42XOR(val) ((keyFrom28To48 & (0x1U << (42 - 28))) ? ~(val) : (val))
#define K43XOR(val) ((keyFrom28To48 & (0x1U << (43 - 28))) ? ~(val) : (val))
#define K44XOR(val) ((keyFrom28To48 & (0x1U << (44 - 28))) ? ~(val) : (val))
#define K45XOR(val) ((keyFrom28To48 & (0x1U << (45 - 28))) ? ~(val) : (val))
#define K46XOR(val) ((keyFrom28To48 & (0x1U << (46 - 28))) ? ~(val) : (val))
#define K47XOR(val) ((keyFrom28To48 & (0x1U << (47 - 28))) ? ~(val) : (val))
#define K48XOR(val) ((keyFrom28To48 & (0x1U << (48 - 28))) ? ~(val) : (val))
	DES_Vector K49 = CUDA_keyFrom49To55Array[0];
	DES_Vector K50 = CUDA_keyFrom49To55Array[1];
	DES_Vector K51 = CUDA_keyFrom49To55Array[2];
	DES_Vector K52 = CUDA_keyFrom49To55Array[3];
	DES_Vector K53 = CUDA_keyFrom49To55Array[4];
	DES_Vector K54 = CUDA_keyFrom49To55Array[5];
	DES_Vector K55 = CUDA_keyFrom49To55Array[6];
#define K49XOR(val) ((val) ^ K49)
#define K50XOR(val) ((val) ^ K50)
#define K51XOR(val) ((val) ^ K51)
#define K52XOR(val) ((val) ^ K52)
#define K53XOR(val) ((val) ^ K53)
#define K54XOR(val) ((val) ^ K54)
#define K55XOR(val) ((val) ^ K55)

#if FALSE

#pragma unroll 1 // Do not unroll.
	for (int i = 0; i < 13; ++i) {
		// ROUND_A(0);
		switch (threadIdx.y) {
		case 0: s1(K12XOR(db[E0]), K46XOR(db[E1]), K33XOR(db[E2]), K52XOR(db[E3]), K48XOR(db[E4]), K20XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K53XOR(db[11*N]), K06XOR(db[12*N]), K31XOR(db[13*N]), K25XOR(db[14*N]), K19XOR(db[15*N]), K41XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K04XOR(db[ 7*N]), K32XOR(db[ 8*N]), K26XOR(db[ 9*N]), K27XOR(db[10*N]), K38XOR(db[11*N]), K54XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K34XOR(db[E0]), K55XOR(db[E1]), K05XOR(db[E2]), K13XOR(db[E3]), K18XOR(db[E4]), K40XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K15XOR(db[E0]), K24XOR(db[E1]), K28XOR(db[E2]), K43XOR(db[E3]), K30XOR(db[E4]), K03XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K37XOR(db[27*N]), K08XOR(db[28*N]), K09XOR(db[29*N]), K50XOR(db[30*N]), K42XOR(db[31*N]), K21XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K51XOR(db[23*N]), K16XOR(db[24*N]), K29XOR(db[25*N]), K49XOR(db[26*N]), K07XOR(db[27*N]), K17XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K35XOR(db[E0]), K22XOR(db[E1]), K02XOR(db[E2]), K44XOR(db[E3]), K14XOR(db[E4]), K23XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(0);
		switch (threadIdx.y) {
		case 0: s1(K05XOR(db[(E0)+(32*N)]), K39XOR(db[(E1)+(32*N)]), K26XOR(db[(E2)+(32*N)]), K45XOR(db[(E3)+(32*N)]), K41XOR(db[(E4)+(32*N)]), K13XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K46XOR(db[43*N]), K54XOR(db[44*N]), K55XOR(db[45*N]), K18XOR(db[46*N]), K12XOR(db[47*N]), K34XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K52XOR(db[39*N]), K25XOR(db[40*N]), K19XOR(db[41*N]), K20XOR(db[42*N]), K31XOR(db[43*N]), K47XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K27XOR(db[(E0)+(32*N)]), K48XOR(db[(E1)+(32*N)]), K53XOR(db[(E2)+(32*N)]), K06XOR(db[(E3)+(32*N)]), K11XOR(db[(E4)+(32*N)]), K33XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K08XOR(db[(E0)+(32*N)]), K17XOR(db[(E1)+(32*N)]), K21XOR(db[(E2)+(32*N)]), K36XOR(db[(E3)+(32*N)]), K23XOR(db[(E4)+(32*N)]), K49XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K30XOR(db[59*N]), K01XOR(db[60*N]), K02XOR(db[61*N]), K43XOR(db[62*N]), K35XOR(db[63*N]), K14XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K44XOR(db[55*N]), K09XOR(db[56*N]), K22XOR(db[57*N]), K42XOR(db[58*N]), K00XOR(db[59*N]), K10XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K28XOR(db[(E0)+(32*N)]), K15XOR(db[(E1)+(32*N)]), K24XOR(db[(E2)+(32*N)]), K37XOR(db[(E3)+(32*N)]), K07XOR(db[(E4)+(32*N)]), K16XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(96);
		switch (threadIdx.y) {
		case 0: s1(K46XOR(db[E0]), K25XOR(db[E1]), K12XOR(db[E2]), K31XOR(db[E3]), K27XOR(db[E4]), K54XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K32XOR(db[11*N]), K40XOR(db[12*N]), K41XOR(db[13*N]), K04XOR(db[14*N]), K53XOR(db[15*N]), K20XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K38XOR(db[ 7*N]), K11XOR(db[ 8*N]), K05XOR(db[ 9*N]), K06XOR(db[10*N]), K48XOR(db[11*N]), K33XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K13XOR(db[E0]), K34XOR(db[E1]), K39XOR(db[E2]), K47XOR(db[E3]), K52XOR(db[E4]), K19XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K51XOR(db[E0]), K03XOR(db[E1]), K07XOR(db[E2]), K22XOR(db[E3]), K09XOR(db[E4]), K35XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K16XOR(db[27*N]), K44XOR(db[28*N]), K17XOR(db[29*N]), K29XOR(db[30*N]), K21XOR(db[31*N]), K00XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K30XOR(db[23*N]), K24XOR(db[24*N]), K08XOR(db[25*N]), K28XOR(db[26*N]), K43XOR(db[27*N]), K49XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K14XOR(db[E0]), K01XOR(db[E1]), K10XOR(db[E2]), K23XOR(db[E3]), K50XOR(db[E4]), K02XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(96);
		switch (threadIdx.y) {
		case 0: s1(K32XOR(db[(E0)+(32*N)]), K11XOR(db[(E1)+(32*N)]), K53XOR(db[(E2)+(32*N)]), K48XOR(db[(E3)+(32*N)]), K13XOR(db[(E4)+(32*N)]), K40XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K18XOR(db[43*N]), K26XOR(db[44*N]), K27XOR(db[45*N]), K45XOR(db[46*N]), K39XOR(db[47*N]), K06XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K55XOR(db[39*N]), K52XOR(db[40*N]), K46XOR(db[41*N]), K47XOR(db[42*N]), K34XOR(db[43*N]), K19XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K54XOR(db[(E0)+(32*N)]), K20XOR(db[(E1)+(32*N)]), K25XOR(db[(E2)+(32*N)]), K33XOR(db[(E3)+(32*N)]), K38XOR(db[(E4)+(32*N)]), K05XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K37XOR(db[(E0)+(32*N)]), K42XOR(db[(E1)+(32*N)]), K50XOR(db[(E2)+(32*N)]), K08XOR(db[(E3)+(32*N)]), K24XOR(db[(E4)+(32*N)]), K21XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K02XOR(db[59*N]), K30XOR(db[60*N]), K03XOR(db[61*N]), K15XOR(db[62*N]), K07XOR(db[63*N]), K43XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K16XOR(db[55*N]), K10XOR(db[56*N]), K51XOR(db[57*N]), K14XOR(db[58*N]), K29XOR(db[59*N]), K35XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K00XOR(db[(E0)+(32*N)]), K44XOR(db[(E1)+(32*N)]), K49XOR(db[(E2)+(32*N)]), K09XOR(db[(E3)+(32*N)]), K36XOR(db[(E4)+(32*N)]), K17XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(192);
		switch (threadIdx.y) {
		case 0: s1(K18XOR(db[E0]), K52XOR(db[E1]), K39XOR(db[E2]), K34XOR(db[E3]), K54XOR(db[E4]), K26XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K04XOR(db[11*N]), K12XOR(db[12*N]), K13XOR(db[13*N]), K31XOR(db[14*N]), K25XOR(db[15*N]), K47XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K41XOR(db[ 7*N]), K38XOR(db[ 8*N]), K32XOR(db[ 9*N]), K33XOR(db[10*N]), K20XOR(db[11*N]), K05XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K40XOR(db[E0]), K06XOR(db[E1]), K11XOR(db[E2]), K19XOR(db[E3]), K55XOR(db[E4]), K46XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K23XOR(db[E0]), K28XOR(db[E1]), K36XOR(db[E2]), K51XOR(db[E3]), K10XOR(db[E4]), K07XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K17XOR(db[27*N]), K16XOR(db[28*N]), K42XOR(db[29*N]), K01XOR(db[30*N]), K50XOR(db[31*N]), K29XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K02XOR(db[23*N]), K49XOR(db[24*N]), K37XOR(db[25*N]), K00XOR(db[26*N]), K15XOR(db[27*N]), K21XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K43XOR(db[E0]), K30XOR(db[E1]), K35XOR(db[E2]), K24XOR(db[E3]), K22XOR(db[E4]), K03XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(192);
		switch (threadIdx.y) {
		case 0: s1(K04XOR(db[(E0)+(32*N)]), K38XOR(db[(E1)+(32*N)]), K25XOR(db[(E2)+(32*N)]), K20XOR(db[(E3)+(32*N)]), K40XOR(db[(E4)+(32*N)]), K12XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K45XOR(db[43*N]), K53XOR(db[44*N]), K54XOR(db[45*N]), K48XOR(db[46*N]), K11XOR(db[47*N]), K33XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K27XOR(db[39*N]), K55XOR(db[40*N]), K18XOR(db[41*N]), K19XOR(db[42*N]), K06XOR(db[43*N]), K46XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K26XOR(db[(E0)+(32*N)]), K47XOR(db[(E1)+(32*N)]), K52XOR(db[(E2)+(32*N)]), K05XOR(db[(E3)+(32*N)]), K41XOR(db[(E4)+(32*N)]), K32XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K09XOR(db[(E0)+(32*N)]), K14XOR(db[(E1)+(32*N)]), K22XOR(db[(E2)+(32*N)]), K37XOR(db[(E3)+(32*N)]), K49XOR(db[(E4)+(32*N)]), K50XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K03XOR(db[59*N]), K02XOR(db[60*N]), K28XOR(db[61*N]), K44XOR(db[62*N]), K36XOR(db[63*N]), K15XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K17XOR(db[55*N]), K35XOR(db[56*N]), K23XOR(db[57*N]), K43XOR(db[58*N]), K01XOR(db[59*N]), K07XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K29XOR(db[(E0)+(32*N)]), K16XOR(db[(E1)+(32*N)]), K21XOR(db[(E2)+(32*N)]), K10XOR(db[(E3)+(32*N)]), K08XOR(db[(E4)+(32*N)]), K42XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(288);
		switch (threadIdx.y) {
		case 0: s1(K45XOR(db[E0]), K55XOR(db[E1]), K11XOR(db[E2]), K06XOR(db[E3]), K26XOR(db[E4]), K53XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K31XOR(db[11*N]), K39XOR(db[12*N]), K40XOR(db[13*N]), K34XOR(db[14*N]), K52XOR(db[15*N]), K19XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K13XOR(db[ 7*N]), K41XOR(db[ 8*N]), K04XOR(db[ 9*N]), K05XOR(db[10*N]), K47XOR(db[11*N]), K32XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K12XOR(db[E0]), K33XOR(db[E1]), K38XOR(db[E2]), K46XOR(db[E3]), K27XOR(db[E4]), K18XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K24XOR(db[E0]), K00XOR(db[E1]), K08XOR(db[E2]), K23XOR(db[E3]), K35XOR(db[E4]), K36XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K42XOR(db[27*N]), K17XOR(db[28*N]), K14XOR(db[29*N]), K30XOR(db[30*N]), K22XOR(db[31*N]), K01XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K03XOR(db[23*N]), K21XOR(db[24*N]), K09XOR(db[25*N]), K29XOR(db[26*N]), K44XOR(db[27*N]), K50XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K15XOR(db[E0]), K02XOR(db[E1]), K07XOR(db[E2]), K49XOR(db[E3]), K51XOR(db[E4]), K28XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(288);
		switch (threadIdx.y) {
		case 0: s1(K31XOR(db[(E0)+(32*N)]), K41XOR(db[(E1)+(32*N)]), K52XOR(db[(E2)+(32*N)]), K47XOR(db[(E3)+(32*N)]), K12XOR(db[(E4)+(32*N)]), K39XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K48XOR(db[43*N]), K25XOR(db[44*N]), K26XOR(db[45*N]), K20XOR(db[46*N]), K38XOR(db[47*N]), K05XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K54XOR(db[39*N]), K27XOR(db[40*N]), K45XOR(db[41*N]), K46XOR(db[42*N]), K33XOR(db[43*N]), K18XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K53XOR(db[(E0)+(32*N)]), K19XOR(db[(E1)+(32*N)]), K55XOR(db[(E2)+(32*N)]), K32XOR(db[(E3)+(32*N)]), K13XOR(db[(E4)+(32*N)]), K04XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K10XOR(db[(E0)+(32*N)]), K43XOR(db[(E1)+(32*N)]), K51XOR(db[(E2)+(32*N)]), K09XOR(db[(E3)+(32*N)]), K21XOR(db[(E4)+(32*N)]), K22XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K28XOR(db[59*N]), K03XOR(db[60*N]), K00XOR(db[61*N]), K16XOR(db[62*N]), K08XOR(db[63*N]), K44XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K42XOR(db[55*N]), K07XOR(db[56*N]), K24XOR(db[57*N]), K15XOR(db[58*N]), K30XOR(db[59*N]), K36XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K01XOR(db[(E0)+(32*N)]), K17XOR(db[(E1)+(32*N)]), K50XOR(db[(E2)+(32*N)]), K35XOR(db[(E3)+(32*N)]), K37XOR(db[(E4)+(32*N)]), K14XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(384);
		switch (threadIdx.y) {
		case 0: s1(K55XOR(db[E0]), K34XOR(db[E1]), K45XOR(db[E2]), K40XOR(db[E3]), K05XOR(db[E4]), K32XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K41XOR(db[11*N]), K18XOR(db[12*N]), K19XOR(db[13*N]), K13XOR(db[14*N]), K31XOR(db[15*N]), K53XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K47XOR(db[ 7*N]), K20XOR(db[ 8*N]), K38XOR(db[ 9*N]), K39XOR(db[10*N]), K26XOR(db[11*N]), K11XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K46XOR(db[E0]), K12XOR(db[E1]), K48XOR(db[E2]), K25XOR(db[E3]), K06XOR(db[E4]), K52XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K03XOR(db[E0]), K36XOR(db[E1]), K44XOR(db[E2]), K02XOR(db[E3]), K14XOR(db[E4]), K15XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K21XOR(db[27*N]), K49XOR(db[28*N]), K50XOR(db[29*N]), K09XOR(db[30*N]), K01XOR(db[31*N]), K37XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K35XOR(db[23*N]), K00XOR(db[24*N]), K17XOR(db[25*N]), K08XOR(db[26*N]), K23XOR(db[27*N]), K29XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K51XOR(db[E0]), K10XOR(db[E1]), K43XOR(db[E2]), K28XOR(db[E3]), K30XOR(db[E4]), K07XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(384);
		switch (threadIdx.y) {
		case 0: s1(K41XOR(db[(E0)+(32*N)]), K20XOR(db[(E1)+(32*N)]), K31XOR(db[(E2)+(32*N)]), K26XOR(db[(E3)+(32*N)]), K46XOR(db[(E4)+(32*N)]), K18XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K27XOR(db[43*N]), K04XOR(db[44*N]), K05XOR(db[45*N]), K54XOR(db[46*N]), K48XOR(db[47*N]), K39XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K33XOR(db[39*N]), K06XOR(db[40*N]), K55XOR(db[41*N]), K25XOR(db[42*N]), K12XOR(db[43*N]), K52XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K32XOR(db[(E0)+(32*N)]), K53XOR(db[(E1)+(32*N)]), K34XOR(db[(E2)+(32*N)]), K11XOR(db[(E3)+(32*N)]), K47XOR(db[(E4)+(32*N)]), K38XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K42XOR(db[(E0)+(32*N)]), K22XOR(db[(E1)+(32*N)]), K30XOR(db[(E2)+(32*N)]), K17XOR(db[(E3)+(32*N)]), K00XOR(db[(E4)+(32*N)]), K01XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K07XOR(db[59*N]), K35XOR(db[60*N]), K36XOR(db[61*N]), K24XOR(db[62*N]), K44XOR(db[63*N]), K23XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K21XOR(db[55*N]), K43XOR(db[56*N]), K03XOR(db[57*N]), K51XOR(db[58*N]), K09XOR(db[59*N]), K15XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K37XOR(db[(E0)+(32*N)]), K49XOR(db[(E1)+(32*N)]), K29XOR(db[(E2)+(32*N)]), K14XOR(db[(E3)+(32*N)]), K16XOR(db[(E4)+(32*N)]), K50XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(480);
		switch (threadIdx.y) {
		case 0: s1(K27XOR(db[E0]), K06XOR(db[E1]), K48XOR(db[E2]), K12XOR(db[E3]), K32XOR(db[E4]), K04XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K13XOR(db[11*N]), K45XOR(db[12*N]), K46XOR(db[13*N]), K40XOR(db[14*N]), K34XOR(db[15*N]), K25XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K19XOR(db[ 7*N]), K47XOR(db[ 8*N]), K41XOR(db[ 9*N]), K11XOR(db[10*N]), K53XOR(db[11*N]), K38XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K18XOR(db[E0]), K39XOR(db[E1]), K20XOR(db[E2]), K52XOR(db[E3]), K33XOR(db[E4]), K55XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K28XOR(db[E0]), K08XOR(db[E1]), K16XOR(db[E2]), K03XOR(db[E3]), K43XOR(db[E4]), K44XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K50XOR(db[27*N]), K21XOR(db[28*N]), K22XOR(db[29*N]), K10XOR(db[30*N]), K30XOR(db[31*N]), K09XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K07XOR(db[23*N]), K29XOR(db[24*N]), K42XOR(db[25*N]), K37XOR(db[26*N]), K24XOR(db[27*N]), K01XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K23XOR(db[E0]), K35XOR(db[E1]), K15XOR(db[E2]), K00XOR(db[E3]), K02XOR(db[E4]), K36XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(480);
		switch (threadIdx.y) {
		case 0: s1(K13XOR(db[(E0)+(32*N)]), K47XOR(db[(E1)+(32*N)]), K34XOR(db[(E2)+(32*N)]), K53XOR(db[(E3)+(32*N)]), K18XOR(db[(E4)+(32*N)]), K45XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K54XOR(db[43*N]), K31XOR(db[44*N]), K32XOR(db[45*N]), K26XOR(db[46*N]), K20XOR(db[47*N]), K11XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K05XOR(db[39*N]), K33XOR(db[40*N]), K27XOR(db[41*N]), K52XOR(db[42*N]), K39XOR(db[43*N]), K55XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K04XOR(db[(E0)+(32*N)]), K25XOR(db[(E1)+(32*N)]), K06XOR(db[(E2)+(32*N)]), K38XOR(db[(E3)+(32*N)]), K19XOR(db[(E4)+(32*N)]), K41XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K14XOR(db[(E0)+(32*N)]), K51XOR(db[(E1)+(32*N)]), K02XOR(db[(E2)+(32*N)]), K42XOR(db[(E3)+(32*N)]), K29XOR(db[(E4)+(32*N)]), K30XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K36XOR(db[59*N]), K07XOR(db[60*N]), K08XOR(db[61*N]), K49XOR(db[62*N]), K16XOR(db[63*N]), K24XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K50XOR(db[55*N]), K15XOR(db[56*N]), K28XOR(db[57*N]), K23XOR(db[58*N]), K10XOR(db[59*N]), K44XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K09XOR(db[(E0)+(32*N)]), K21XOR(db[(E1)+(32*N)]), K01XOR(db[(E2)+(32*N)]), K43XOR(db[(E3)+(32*N)]), K17XOR(db[(E4)+(32*N)]), K22XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(576);
		switch (threadIdx.y) {
		case 0: s1(K54XOR(db[E0]), K33XOR(db[E1]), K20XOR(db[E2]), K39XOR(db[E3]), K04XOR(db[E4]), K31XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K40XOR(db[11*N]), K48XOR(db[12*N]), K18XOR(db[13*N]), K12XOR(db[14*N]), K06XOR(db[15*N]), K52XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K46XOR(db[ 7*N]), K19XOR(db[ 8*N]), K13XOR(db[ 9*N]), K38XOR(db[10*N]), K25XOR(db[11*N]), K41XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K45XOR(db[E0]), K11XOR(db[E1]), K47XOR(db[E2]), K55XOR(db[E3]), K05XOR(db[E4]), K27XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K00XOR(db[E0]), K37XOR(db[E1]), K17XOR(db[E2]), K28XOR(db[E3]), K15XOR(db[E4]), K16XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K22XOR(db[27*N]), K50XOR(db[28*N]), K51XOR(db[29*N]), K35XOR(db[30*N]), K02XOR(db[31*N]), K10XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K36XOR(db[23*N]), K01XOR(db[24*N]), K14XOR(db[25*N]), K09XOR(db[26*N]), K49XOR(db[27*N]), K30XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K24XOR(db[E0]), K07XOR(db[E1]), K44XOR(db[E2]), K29XOR(db[E3]), K03XOR(db[E4]), K08XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(576);
		switch (threadIdx.y) {
		case 0: s1(K40XOR(db[(E0)+(32*N)]), K19XOR(db[(E1)+(32*N)]), K06XOR(db[(E2)+(32*N)]), K25XOR(db[(E3)+(32*N)]), K45XOR(db[(E4)+(32*N)]), K48XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K26XOR(db[43*N]), K34XOR(db[44*N]), K04XOR(db[45*N]), K53XOR(db[46*N]), K47XOR(db[47*N]), K38XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K32XOR(db[39*N]), K05XOR(db[40*N]), K54XOR(db[41*N]), K55XOR(db[42*N]), K11XOR(db[43*N]), K27XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K31XOR(db[(E0)+(32*N)]), K52XOR(db[(E1)+(32*N)]), K33XOR(db[(E2)+(32*N)]), K41XOR(db[(E3)+(32*N)]), K46XOR(db[(E4)+(32*N)]), K13XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K43XOR(db[(E0)+(32*N)]), K23XOR(db[(E1)+(32*N)]), K03XOR(db[(E2)+(32*N)]), K14XOR(db[(E3)+(32*N)]), K01XOR(db[(E4)+(32*N)]), K02XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K08XOR(db[59*N]), K36XOR(db[60*N]), K37XOR(db[61*N]), K21XOR(db[62*N]), K17XOR(db[63*N]), K49XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K22XOR(db[55*N]), K44XOR(db[56*N]), K00XOR(db[57*N]), K24XOR(db[58*N]), K35XOR(db[59*N]), K16XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K10XOR(db[(E0)+(32*N)]), K50XOR(db[(E1)+(32*N)]), K30XOR(db[(E2)+(32*N)]), K15XOR(db[(E3)+(32*N)]), K42XOR(db[(E4)+(32*N)]), K51XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(672);
		switch (threadIdx.y) {
		case 0: s1(K26XOR(db[E0]), K05XOR(db[E1]), K47XOR(db[E2]), K11XOR(db[E3]), K31XOR(db[E4]), K34XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K12XOR(db[11*N]), K20XOR(db[12*N]), K45XOR(db[13*N]), K39XOR(db[14*N]), K33XOR(db[15*N]), K55XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K18XOR(db[ 7*N]), K46XOR(db[ 8*N]), K40XOR(db[ 9*N]), K41XOR(db[10*N]), K52XOR(db[11*N]), K13XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K48XOR(db[E0]), K38XOR(db[E1]), K19XOR(db[E2]), K27XOR(db[E3]), K32XOR(db[E4]), K54XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K29XOR(db[E0]), K09XOR(db[E1]), K42XOR(db[E2]), K00XOR(db[E3]), K44XOR(db[E4]), K17XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K51XOR(db[27*N]), K22XOR(db[28*N]), K23XOR(db[29*N]), K07XOR(db[30*N]), K03XOR(db[31*N]), K35XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K08XOR(db[23*N]), K30XOR(db[24*N]), K43XOR(db[25*N]), K10XOR(db[26*N]), K21XOR(db[27*N]), K02XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K49XOR(db[E0]), K36XOR(db[E1]), K16XOR(db[E2]), K01XOR(db[E3]), K28XOR(db[E4]), K37XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(672);
		switch (threadIdx.y) {
		case 0: s1(K19XOR(db[(E0)+(32*N)]), K53XOR(db[(E1)+(32*N)]), K40XOR(db[(E2)+(32*N)]), K04XOR(db[(E3)+(32*N)]), K55XOR(db[(E4)+(32*N)]), K27XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K05XOR(db[43*N]), K13XOR(db[44*N]), K38XOR(db[45*N]), K32XOR(db[46*N]), K26XOR(db[47*N]), K48XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K11XOR(db[39*N]), K39XOR(db[40*N]), K33XOR(db[41*N]), K34XOR(db[42*N]), K45XOR(db[43*N]), K06XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K41XOR(db[(E0)+(32*N)]), K31XOR(db[(E1)+(32*N)]), K12XOR(db[(E2)+(32*N)]), K20XOR(db[(E3)+(32*N)]), K25XOR(db[(E4)+(32*N)]), K47XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K22XOR(db[(E0)+(32*N)]), K02XOR(db[(E1)+(32*N)]), K35XOR(db[(E2)+(32*N)]), K50XOR(db[(E3)+(32*N)]), K37XOR(db[(E4)+(32*N)]), K10XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K44XOR(db[59*N]), K15XOR(db[60*N]), K16XOR(db[61*N]), K00XOR(db[62*N]), K49XOR(db[63*N]), K28XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K01XOR(db[55*N]), K23XOR(db[56*N]), K36XOR(db[57*N]), K03XOR(db[58*N]), K14XOR(db[59*N]), K24XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K42XOR(db[(E0)+(32*N)]), K29XOR(db[(E1)+(32*N)]), K09XOR(db[(E2)+(32*N)]), K51XOR(db[(E3)+(32*N)]), K21XOR(db[(E4)+(32*N)]), K30XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		if (i >= 12)
			break;

		// ROUND_B(-48);
		switch (threadIdx.y) {
		case 0: s1(K12XOR(db[(E0)+(32*N)]), K46XOR(db[(E1)+(32*N)]), K33XOR(db[(E2)+(32*N)]), K52XOR(db[(E3)+(32*N)]), K48XOR(db[(E4)+(32*N)]), K20XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K53XOR(db[43*N]), K06XOR(db[44*N]), K31XOR(db[45*N]), K25XOR(db[46*N]), K19XOR(db[47*N]), K41XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K04XOR(db[39*N]), K32XOR(db[40*N]), K26XOR(db[41*N]), K27XOR(db[42*N]), K38XOR(db[43*N]), K54XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K34XOR(db[(E0)+(32*N)]), K55XOR(db[(E1)+(32*N)]), K05XOR(db[(E2)+(32*N)]), K13XOR(db[(E3)+(32*N)]), K18XOR(db[(E4)+(32*N)]), K40XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K15XOR(db[(E0)+(32*N)]), K24XOR(db[(E1)+(32*N)]), K28XOR(db[(E2)+(32*N)]), K43XOR(db[(E3)+(32*N)]), K30XOR(db[(E4)+(32*N)]), K03XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K37XOR(db[59*N]), K08XOR(db[60*N]), K09XOR(db[61*N]), K50XOR(db[62*N]), K42XOR(db[63*N]), K21XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K51XOR(db[55*N]), K16XOR(db[56*N]), K29XOR(db[57*N]), K49XOR(db[58*N]), K07XOR(db[59*N]), K17XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K35XOR(db[(E0)+(32*N)]), K22XOR(db[(E1)+(32*N)]), K02XOR(db[(E2)+(32*N)]), K44XOR(db[(E3)+(32*N)]), K14XOR(db[(E4)+(32*N)]), K23XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(48);
		switch (threadIdx.y) {
		case 0: s1(K05XOR(db[E0]), K39XOR(db[E1]), K26XOR(db[E2]), K45XOR(db[E3]), K41XOR(db[E4]), K13XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K46XOR(db[11*N]), K54XOR(db[12*N]), K55XOR(db[13*N]), K18XOR(db[14*N]), K12XOR(db[15*N]), K34XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K52XOR(db[ 7*N]), K25XOR(db[ 8*N]), K19XOR(db[ 9*N]), K20XOR(db[10*N]), K31XOR(db[11*N]), K47XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K27XOR(db[E0]), K48XOR(db[E1]), K53XOR(db[E2]), K06XOR(db[E3]), K11XOR(db[E4]), K33XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K08XOR(db[E0]), K17XOR(db[E1]), K21XOR(db[E2]), K36XOR(db[E3]), K23XOR(db[E4]), K49XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K30XOR(db[27*N]), K01XOR(db[28*N]), K02XOR(db[29*N]), K43XOR(db[30*N]), K35XOR(db[31*N]), K14XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K44XOR(db[23*N]), K09XOR(db[24*N]), K22XOR(db[25*N]), K42XOR(db[26*N]), K00XOR(db[27*N]), K10XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K28XOR(db[E0]), K15XOR(db[E1]), K24XOR(db[E2]), K37XOR(db[E3]), K07XOR(db[E4]), K16XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(48);
		switch (threadIdx.y) {
		case 0: s1(K46XOR(db[(E0)+(32*N)]), K25XOR(db[(E1)+(32*N)]), K12XOR(db[(E2)+(32*N)]), K31XOR(db[(E3)+(32*N)]), K27XOR(db[(E4)+(32*N)]), K54XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K32XOR(db[43*N]), K40XOR(db[44*N]), K41XOR(db[45*N]), K04XOR(db[46*N]), K53XOR(db[47*N]), K20XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K38XOR(db[39*N]), K11XOR(db[40*N]), K05XOR(db[41*N]), K06XOR(db[42*N]), K48XOR(db[43*N]), K33XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K13XOR(db[(E0)+(32*N)]), K34XOR(db[(E1)+(32*N)]), K39XOR(db[(E2)+(32*N)]), K47XOR(db[(E3)+(32*N)]), K52XOR(db[(E4)+(32*N)]), K19XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K51XOR(db[(E0)+(32*N)]), K03XOR(db[(E1)+(32*N)]), K07XOR(db[(E2)+(32*N)]), K22XOR(db[(E3)+(32*N)]), K09XOR(db[(E4)+(32*N)]), K35XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K16XOR(db[59*N]), K44XOR(db[60*N]), K17XOR(db[61*N]), K29XOR(db[62*N]), K21XOR(db[63*N]), K00XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K30XOR(db[55*N]), K24XOR(db[56*N]), K08XOR(db[57*N]), K28XOR(db[58*N]), K43XOR(db[59*N]), K49XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K14XOR(db[(E0)+(32*N)]), K01XOR(db[(E1)+(32*N)]), K10XOR(db[(E2)+(32*N)]), K23XOR(db[(E3)+(32*N)]), K50XOR(db[(E4)+(32*N)]), K02XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(144);
		switch (threadIdx.y) {
		case 0: s1(K32XOR(db[E0]), K11XOR(db[E1]), K53XOR(db[E2]), K48XOR(db[E3]), K13XOR(db[E4]), K40XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K18XOR(db[11*N]), K26XOR(db[12*N]), K27XOR(db[13*N]), K45XOR(db[14*N]), K39XOR(db[15*N]), K06XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K55XOR(db[ 7*N]), K52XOR(db[ 8*N]), K46XOR(db[ 9*N]), K47XOR(db[10*N]), K34XOR(db[11*N]), K19XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K54XOR(db[E0]), K20XOR(db[E1]), K25XOR(db[E2]), K33XOR(db[E3]), K38XOR(db[E4]), K05XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K37XOR(db[E0]), K42XOR(db[E1]), K50XOR(db[E2]), K08XOR(db[E3]), K24XOR(db[E4]), K21XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K02XOR(db[27*N]), K30XOR(db[28*N]), K03XOR(db[29*N]), K15XOR(db[30*N]), K07XOR(db[31*N]), K43XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K16XOR(db[23*N]), K10XOR(db[24*N]), K51XOR(db[25*N]), K14XOR(db[26*N]), K29XOR(db[27*N]), K35XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K00XOR(db[E0]), K44XOR(db[E1]), K49XOR(db[E2]), K09XOR(db[E3]), K36XOR(db[E4]), K17XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(144);
		switch (threadIdx.y) {
		case 0: s1(K18XOR(db[(E0)+(32*N)]), K52XOR(db[(E1)+(32*N)]), K39XOR(db[(E2)+(32*N)]), K34XOR(db[(E3)+(32*N)]), K54XOR(db[(E4)+(32*N)]), K26XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K04XOR(db[43*N]), K12XOR(db[44*N]), K13XOR(db[45*N]), K31XOR(db[46*N]), K25XOR(db[47*N]), K47XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K41XOR(db[39*N]), K38XOR(db[40*N]), K32XOR(db[41*N]), K33XOR(db[42*N]), K20XOR(db[43*N]), K05XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K40XOR(db[(E0)+(32*N)]), K06XOR(db[(E1)+(32*N)]), K11XOR(db[(E2)+(32*N)]), K19XOR(db[(E3)+(32*N)]), K55XOR(db[(E4)+(32*N)]), K46XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K23XOR(db[(E0)+(32*N)]), K28XOR(db[(E1)+(32*N)]), K36XOR(db[(E2)+(32*N)]), K51XOR(db[(E3)+(32*N)]), K10XOR(db[(E4)+(32*N)]), K07XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K17XOR(db[59*N]), K16XOR(db[60*N]), K42XOR(db[61*N]), K01XOR(db[62*N]), K50XOR(db[63*N]), K29XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K02XOR(db[55*N]), K49XOR(db[56*N]), K37XOR(db[57*N]), K00XOR(db[58*N]), K15XOR(db[59*N]), K21XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K43XOR(db[(E0)+(32*N)]), K30XOR(db[(E1)+(32*N)]), K35XOR(db[(E2)+(32*N)]), K24XOR(db[(E3)+(32*N)]), K22XOR(db[(E4)+(32*N)]), K03XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(240);
		switch (threadIdx.y) {
		case 0: s1(K04XOR(db[E0]), K38XOR(db[E1]), K25XOR(db[E2]), K20XOR(db[E3]), K40XOR(db[E4]), K12XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K45XOR(db[11*N]), K53XOR(db[12*N]), K54XOR(db[13*N]), K48XOR(db[14*N]), K11XOR(db[15*N]), K33XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K27XOR(db[ 7*N]), K55XOR(db[ 8*N]), K18XOR(db[ 9*N]), K19XOR(db[10*N]), K06XOR(db[11*N]), K46XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K26XOR(db[E0]), K47XOR(db[E1]), K52XOR(db[E2]), K05XOR(db[E3]), K41XOR(db[E4]), K32XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K09XOR(db[E0]), K14XOR(db[E1]), K22XOR(db[E2]), K37XOR(db[E3]), K49XOR(db[E4]), K50XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K03XOR(db[27*N]), K02XOR(db[28*N]), K28XOR(db[29*N]), K44XOR(db[30*N]), K36XOR(db[31*N]), K15XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K17XOR(db[23*N]), K35XOR(db[24*N]), K23XOR(db[25*N]), K43XOR(db[26*N]), K01XOR(db[27*N]), K07XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K29XOR(db[E0]), K16XOR(db[E1]), K21XOR(db[E2]), K10XOR(db[E3]), K08XOR(db[E4]), K42XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(240);
		switch (threadIdx.y) {
		case 0: s1(K45XOR(db[(E0)+(32*N)]), K55XOR(db[(E1)+(32*N)]), K11XOR(db[(E2)+(32*N)]), K06XOR(db[(E3)+(32*N)]), K26XOR(db[(E4)+(32*N)]), K53XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K31XOR(db[43*N]), K39XOR(db[44*N]), K40XOR(db[45*N]), K34XOR(db[46*N]), K52XOR(db[47*N]), K19XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K13XOR(db[39*N]), K41XOR(db[40*N]), K04XOR(db[41*N]), K05XOR(db[42*N]), K47XOR(db[43*N]), K32XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K12XOR(db[(E0)+(32*N)]), K33XOR(db[(E1)+(32*N)]), K38XOR(db[(E2)+(32*N)]), K46XOR(db[(E3)+(32*N)]), K27XOR(db[(E4)+(32*N)]), K18XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K24XOR(db[(E0)+(32*N)]), K00XOR(db[(E1)+(32*N)]), K08XOR(db[(E2)+(32*N)]), K23XOR(db[(E3)+(32*N)]), K35XOR(db[(E4)+(32*N)]), K36XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K42XOR(db[59*N]), K17XOR(db[60*N]), K14XOR(db[61*N]), K30XOR(db[62*N]), K22XOR(db[63*N]), K01XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K03XOR(db[55*N]), K21XOR(db[56*N]), K09XOR(db[57*N]), K29XOR(db[58*N]), K44XOR(db[59*N]), K50XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K15XOR(db[(E0)+(32*N)]), K02XOR(db[(E1)+(32*N)]), K07XOR(db[(E2)+(32*N)]), K49XOR(db[(E3)+(32*N)]), K51XOR(db[(E4)+(32*N)]), K28XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(336);
		switch (threadIdx.y) {
		case 0: s1(K31XOR(db[E0]), K41XOR(db[E1]), K52XOR(db[E2]), K47XOR(db[E3]), K12XOR(db[E4]), K39XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K48XOR(db[11*N]), K25XOR(db[12*N]), K26XOR(db[13*N]), K20XOR(db[14*N]), K38XOR(db[15*N]), K05XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K54XOR(db[ 7*N]), K27XOR(db[ 8*N]), K45XOR(db[ 9*N]), K46XOR(db[10*N]), K33XOR(db[11*N]), K18XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K53XOR(db[E0]), K19XOR(db[E1]), K55XOR(db[E2]), K32XOR(db[E3]), K13XOR(db[E4]), K04XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K10XOR(db[E0]), K43XOR(db[E1]), K51XOR(db[E2]), K09XOR(db[E3]), K21XOR(db[E4]), K22XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K28XOR(db[27*N]), K03XOR(db[28*N]), K00XOR(db[29*N]), K16XOR(db[30*N]), K08XOR(db[31*N]), K44XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K42XOR(db[23*N]), K07XOR(db[24*N]), K24XOR(db[25*N]), K15XOR(db[26*N]), K30XOR(db[27*N]), K36XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K01XOR(db[E0]), K17XOR(db[E1]), K50XOR(db[E2]), K35XOR(db[E3]), K37XOR(db[E4]), K14XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(336);
		switch (threadIdx.y) {
		case 0: s1(K55XOR(db[(E0)+(32*N)]), K34XOR(db[(E1)+(32*N)]), K45XOR(db[(E2)+(32*N)]), K40XOR(db[(E3)+(32*N)]), K05XOR(db[(E4)+(32*N)]), K32XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K41XOR(db[43*N]), K18XOR(db[44*N]), K19XOR(db[45*N]), K13XOR(db[46*N]), K31XOR(db[47*N]), K53XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K47XOR(db[39*N]), K20XOR(db[40*N]), K38XOR(db[41*N]), K39XOR(db[42*N]), K26XOR(db[43*N]), K11XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K46XOR(db[(E0)+(32*N)]), K12XOR(db[(E1)+(32*N)]), K48XOR(db[(E2)+(32*N)]), K25XOR(db[(E3)+(32*N)]), K06XOR(db[(E4)+(32*N)]), K52XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K03XOR(db[(E0)+(32*N)]), K36XOR(db[(E1)+(32*N)]), K44XOR(db[(E2)+(32*N)]), K02XOR(db[(E3)+(32*N)]), K14XOR(db[(E4)+(32*N)]), K15XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K21XOR(db[59*N]), K49XOR(db[60*N]), K50XOR(db[61*N]), K09XOR(db[62*N]), K01XOR(db[63*N]), K37XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K35XOR(db[55*N]), K00XOR(db[56*N]), K17XOR(db[57*N]), K08XOR(db[58*N]), K23XOR(db[59*N]), K29XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K51XOR(db[(E0)+(32*N)]), K10XOR(db[(E1)+(32*N)]), K43XOR(db[(E2)+(32*N)]), K28XOR(db[(E3)+(32*N)]), K30XOR(db[(E4)+(32*N)]), K07XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(432);
		switch (threadIdx.y) {
		case 0: s1(K41XOR(db[E0]), K20XOR(db[E1]), K31XOR(db[E2]), K26XOR(db[E3]), K46XOR(db[E4]), K18XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K27XOR(db[11*N]), K04XOR(db[12*N]), K05XOR(db[13*N]), K54XOR(db[14*N]), K48XOR(db[15*N]), K39XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K33XOR(db[ 7*N]), K06XOR(db[ 8*N]), K55XOR(db[ 9*N]), K25XOR(db[10*N]), K12XOR(db[11*N]), K52XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K32XOR(db[E0]), K53XOR(db[E1]), K34XOR(db[E2]), K11XOR(db[E3]), K47XOR(db[E4]), K38XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K42XOR(db[E0]), K22XOR(db[E1]), K30XOR(db[E2]), K17XOR(db[E3]), K00XOR(db[E4]), K01XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K07XOR(db[27*N]), K35XOR(db[28*N]), K36XOR(db[29*N]), K24XOR(db[30*N]), K44XOR(db[31*N]), K23XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K21XOR(db[23*N]), K43XOR(db[24*N]), K03XOR(db[25*N]), K51XOR(db[26*N]), K09XOR(db[27*N]), K15XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K37XOR(db[E0]), K49XOR(db[E1]), K29XOR(db[E2]), K14XOR(db[E3]), K16XOR(db[E4]), K50XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(432);
		switch (threadIdx.y) {
		case 0: s1(K27XOR(db[(E0)+(32*N)]), K06XOR(db[(E1)+(32*N)]), K48XOR(db[(E2)+(32*N)]), K12XOR(db[(E3)+(32*N)]), K32XOR(db[(E4)+(32*N)]), K04XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K13XOR(db[43*N]), K45XOR(db[44*N]), K46XOR(db[45*N]), K40XOR(db[46*N]), K34XOR(db[47*N]), K25XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K19XOR(db[39*N]), K47XOR(db[40*N]), K41XOR(db[41*N]), K11XOR(db[42*N]), K53XOR(db[43*N]), K38XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K18XOR(db[(E0)+(32*N)]), K39XOR(db[(E1)+(32*N)]), K20XOR(db[(E2)+(32*N)]), K52XOR(db[(E3)+(32*N)]), K33XOR(db[(E4)+(32*N)]), K55XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K28XOR(db[(E0)+(32*N)]), K08XOR(db[(E1)+(32*N)]), K16XOR(db[(E2)+(32*N)]), K03XOR(db[(E3)+(32*N)]), K43XOR(db[(E4)+(32*N)]), K44XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K50XOR(db[59*N]), K21XOR(db[60*N]), K22XOR(db[61*N]), K10XOR(db[62*N]), K30XOR(db[63*N]), K09XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K07XOR(db[55*N]), K29XOR(db[56*N]), K42XOR(db[57*N]), K37XOR(db[58*N]), K24XOR(db[59*N]), K01XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K23XOR(db[(E0)+(32*N)]), K35XOR(db[(E1)+(32*N)]), K15XOR(db[(E2)+(32*N)]), K00XOR(db[(E3)+(32*N)]), K02XOR(db[(E4)+(32*N)]), K36XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(528);
		switch (threadIdx.y) {
		case 0: s1(K13XOR(db[E0]), K47XOR(db[E1]), K34XOR(db[E2]), K53XOR(db[E3]), K18XOR(db[E4]), K45XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K54XOR(db[11*N]), K31XOR(db[12*N]), K32XOR(db[13*N]), K26XOR(db[14*N]), K20XOR(db[15*N]), K11XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K05XOR(db[ 7*N]), K33XOR(db[ 8*N]), K27XOR(db[ 9*N]), K52XOR(db[10*N]), K39XOR(db[11*N]), K55XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K04XOR(db[E0]), K25XOR(db[E1]), K06XOR(db[E2]), K38XOR(db[E3]), K19XOR(db[E4]), K41XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K14XOR(db[E0]), K51XOR(db[E1]), K02XOR(db[E2]), K42XOR(db[E3]), K29XOR(db[E4]), K30XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K36XOR(db[27*N]), K07XOR(db[28*N]), K08XOR(db[29*N]), K49XOR(db[30*N]), K16XOR(db[31*N]), K24XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K50XOR(db[23*N]), K15XOR(db[24*N]), K28XOR(db[25*N]), K23XOR(db[26*N]), K10XOR(db[27*N]), K44XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K09XOR(db[E0]), K21XOR(db[E1]), K01XOR(db[E2]), K43XOR(db[E3]), K17XOR(db[E4]), K22XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(528);
		switch (threadIdx.y) {
		case 0: s1(K54XOR(db[(E0)+(32*N)]), K33XOR(db[(E1)+(32*N)]), K20XOR(db[(E2)+(32*N)]), K39XOR(db[(E3)+(32*N)]), K04XOR(db[(E4)+(32*N)]), K31XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K40XOR(db[43*N]), K48XOR(db[44*N]), K18XOR(db[45*N]), K12XOR(db[46*N]), K06XOR(db[47*N]), K52XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K46XOR(db[39*N]), K19XOR(db[40*N]), K13XOR(db[41*N]), K38XOR(db[42*N]), K25XOR(db[43*N]), K41XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K45XOR(db[(E0)+(32*N)]), K11XOR(db[(E1)+(32*N)]), K47XOR(db[(E2)+(32*N)]), K55XOR(db[(E3)+(32*N)]), K05XOR(db[(E4)+(32*N)]), K27XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K00XOR(db[(E0)+(32*N)]), K37XOR(db[(E1)+(32*N)]), K17XOR(db[(E2)+(32*N)]), K28XOR(db[(E3)+(32*N)]), K15XOR(db[(E4)+(32*N)]), K16XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K22XOR(db[59*N]), K50XOR(db[60*N]), K51XOR(db[61*N]), K35XOR(db[62*N]), K02XOR(db[63*N]), K10XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K36XOR(db[55*N]), K01XOR(db[56*N]), K14XOR(db[57*N]), K09XOR(db[58*N]), K49XOR(db[59*N]), K30XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K24XOR(db[(E0)+(32*N)]), K07XOR(db[(E1)+(32*N)]), K44XOR(db[(E2)+(32*N)]), K29XOR(db[(E3)+(32*N)]), K03XOR(db[(E4)+(32*N)]), K08XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(624);
		switch (threadIdx.y) {
		case 0: s1(K40XOR(db[E0]), K19XOR(db[E1]), K06XOR(db[E2]), K25XOR(db[E3]), K45XOR(db[E4]), K48XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K26XOR(db[11*N]), K34XOR(db[12*N]), K04XOR(db[13*N]), K53XOR(db[14*N]), K47XOR(db[15*N]), K38XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K32XOR(db[ 7*N]), K05XOR(db[ 8*N]), K54XOR(db[ 9*N]), K55XOR(db[10*N]), K11XOR(db[11*N]), K27XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K31XOR(db[E0]), K52XOR(db[E1]), K33XOR(db[E2]), K41XOR(db[E3]), K46XOR(db[E4]), K13XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K43XOR(db[E0]), K23XOR(db[E1]), K03XOR(db[E2]), K14XOR(db[E3]), K01XOR(db[E4]), K02XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K08XOR(db[27*N]), K36XOR(db[28*N]), K37XOR(db[29*N]), K21XOR(db[30*N]), K17XOR(db[31*N]), K49XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K22XOR(db[23*N]), K44XOR(db[24*N]), K00XOR(db[25*N]), K24XOR(db[26*N]), K35XOR(db[27*N]), K16XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K10XOR(db[E0]), K50XOR(db[E1]), K30XOR(db[E2]), K15XOR(db[E3]), K42XOR(db[E4]), K51XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();

		// ROUND_B(624);
		switch (threadIdx.y) {
		case 0: s1(K26XOR(db[(E0)+(32*N)]), K05XOR(db[(E1)+(32*N)]), K47XOR(db[(E2)+(32*N)]), K11XOR(db[(E3)+(32*N)]), K31XOR(db[(E4)+(32*N)]), K34XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		        s4(K12XOR(db[43*N]), K20XOR(db[44*N]), K45XOR(db[45*N]), K39XOR(db[46*N]), K33XOR(db[47*N]), K55XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); break;
		case 1: s3(K18XOR(db[39*N]), K46XOR(db[40*N]), K40XOR(db[41*N]), K41XOR(db[42*N]), K52XOR(db[43*N]), K13XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		        s2(K48XOR(db[(E0)+(32*N)]), K38XOR(db[(E1)+(32*N)]), K19XOR(db[(E2)+(32*N)]), K27XOR(db[(E3)+(32*N)]), K32XOR(db[(E4)+(32*N)]), K54XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); break;
		case 2: s5(K29XOR(db[(E0)+(32*N)]), K09XOR(db[(E1)+(32*N)]), K42XOR(db[(E2)+(32*N)]), K00XOR(db[(E3)+(32*N)]), K44XOR(db[(E4)+(32*N)]), K17XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		        s8(K51XOR(db[59*N]), K22XOR(db[60*N]), K23XOR(db[61*N]), K07XOR(db[62*N]), K03XOR(db[63*N]), K35XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); break;
		case 3: s7(K08XOR(db[55*N]), K30XOR(db[56*N]), K43XOR(db[57*N]), K10XOR(db[58*N]), K21XOR(db[59*N]), K02XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
		        s6(K49XOR(db[(E0)+(32*N)]), K36XOR(db[(E1)+(32*N)]), K16XOR(db[(E2)+(32*N)]), K01XOR(db[(E3)+(32*N)]), K28XOR(db[(E4)+(32*N)]), K37XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); break;
		}
		__syncthreads();

		// ROUND_A(720);
		switch (threadIdx.y) {
		case 0: s1(K19XOR(db[E0]), K53XOR(db[E1]), K40XOR(db[E2]), K04XOR(db[E3]), K55XOR(db[E4]), K27XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		        s4(K05XOR(db[11*N]), K13XOR(db[12*N]), K38XOR(db[13*N]), K32XOR(db[14*N]), K26XOR(db[15*N]), K48XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); break;
		case 1: s3(K11XOR(db[ 7*N]), K39XOR(db[ 8*N]), K33XOR(db[ 9*N]), K34XOR(db[10*N]), K45XOR(db[11*N]), K06XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		        s2(K41XOR(db[E0]), K31XOR(db[E1]), K12XOR(db[E2]), K20XOR(db[E3]), K25XOR(db[E4]), K47XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); break;
		case 2: s5(K22XOR(db[E0]), K02XOR(db[E1]), K35XOR(db[E2]), K50XOR(db[E3]), K37XOR(db[E4]), K10XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		        s8(K44XOR(db[27*N]), K15XOR(db[28*N]), K16XOR(db[29*N]), K00XOR(db[30*N]), K49XOR(db[31*N]), K28XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); break;
		case 3: s7(K01XOR(db[23*N]), K23XOR(db[24*N]), K36XOR(db[25*N]), K03XOR(db[26*N]), K14XOR(db[27*N]), K24XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		        s6(K42XOR(db[E0]), K29XOR(db[E1]), K09XOR(db[E2]), K51XOR(db[E3]), K21XOR(db[E4]), K30XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); break;
		}
		__syncthreads();
	}

#else

	// For some reason, this routine works better on GTX580.
#pragma unroll 1 // Do not unroll.
	for (int i = 0; i < 13; ++i) {
		switch (threadIdx.y) {
		case 0: 
			s1(K12XOR(db[E0]), K46XOR(db[E1]), K33XOR(db[E2]), K52XOR(db[E3]), K48XOR(db[E4]), K20XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		    s4(K53XOR(db[11*N]), K06XOR(db[12*N]), K31XOR(db[13*N]), K25XOR(db[14*N]), K19XOR(db[15*N]), K41XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K05XOR(db[(E0)+(32*N)]), K39XOR(db[(E1)+(32*N)]), K26XOR(db[(E2)+(32*N)]), K45XOR(db[(E3)+(32*N)]), K41XOR(db[(E4)+(32*N)]), K13XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		    s4(K46XOR(db[43*N]), K54XOR(db[44*N]), K55XOR(db[45*N]), K18XOR(db[46*N]), K12XOR(db[47*N]), K34XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]);
			__syncthreads();
			s1(K46XOR(db[E0]), K25XOR(db[E1]), K12XOR(db[E2]), K31XOR(db[E3]), K27XOR(db[E4]), K54XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
	        s4(K32XOR(db[11*N]), K40XOR(db[12*N]), K41XOR(db[13*N]), K04XOR(db[14*N]), K53XOR(db[15*N]), K20XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]);
			__syncthreads();
			s1(K32XOR(db[(E0)+(32*N)]), K11XOR(db[(E1)+(32*N)]), K53XOR(db[(E2)+(32*N)]), K48XOR(db[(E3)+(32*N)]), K13XOR(db[(E4)+(32*N)]), K40XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
		    s4(K18XOR(db[43*N]), K26XOR(db[44*N]), K27XOR(db[45*N]), K45XOR(db[46*N]), K39XOR(db[47*N]), K06XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]);
			__syncthreads();
			s1(K18XOR(db[E0]), K52XOR(db[E1]), K39XOR(db[E2]), K34XOR(db[E3]), K54XOR(db[E4]), K26XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
		    s4(K04XOR(db[11*N]), K12XOR(db[12*N]), K13XOR(db[13*N]), K31XOR(db[14*N]), K25XOR(db[15*N]), K47XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]);
			__syncthreads();
			s1(K04XOR(db[(E0)+(32*N)]), K38XOR(db[(E1)+(32*N)]), K25XOR(db[(E2)+(32*N)]), K20XOR(db[(E3)+(32*N)]), K40XOR(db[(E4)+(32*N)]), K12XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K45XOR(db[43*N]), K53XOR(db[44*N]), K54XOR(db[45*N]), K48XOR(db[46*N]), K11XOR(db[47*N]), K33XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K45XOR(db[E0]), K55XOR(db[E1]), K11XOR(db[E2]), K06XOR(db[E3]), K26XOR(db[E4]), K53XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K31XOR(db[11*N]), K39XOR(db[12*N]), K40XOR(db[13*N]), K34XOR(db[14*N]), K52XOR(db[15*N]), K19XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K31XOR(db[(E0)+(32*N)]), K41XOR(db[(E1)+(32*N)]), K52XOR(db[(E2)+(32*N)]), K47XOR(db[(E3)+(32*N)]), K12XOR(db[(E4)+(32*N)]), K39XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K48XOR(db[43*N]), K25XOR(db[44*N]), K26XOR(db[45*N]), K20XOR(db[46*N]), K38XOR(db[47*N]), K05XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K55XOR(db[E0]), K34XOR(db[E1]), K45XOR(db[E2]), K40XOR(db[E3]), K05XOR(db[E4]), K32XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K41XOR(db[11*N]), K18XOR(db[12*N]), K19XOR(db[13*N]), K13XOR(db[14*N]), K31XOR(db[15*N]), K53XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K41XOR(db[(E0)+(32*N)]), K20XOR(db[(E1)+(32*N)]), K31XOR(db[(E2)+(32*N)]), K26XOR(db[(E3)+(32*N)]), K46XOR(db[(E4)+(32*N)]), K18XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K27XOR(db[43*N]), K04XOR(db[44*N]), K05XOR(db[45*N]), K54XOR(db[46*N]), K48XOR(db[47*N]), K39XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K27XOR(db[E0]), K06XOR(db[E1]), K48XOR(db[E2]), K12XOR(db[E3]), K32XOR(db[E4]), K04XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K13XOR(db[11*N]), K45XOR(db[12*N]), K46XOR(db[13*N]), K40XOR(db[14*N]), K34XOR(db[15*N]), K25XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K13XOR(db[(E0)+(32*N)]), K47XOR(db[(E1)+(32*N)]), K34XOR(db[(E2)+(32*N)]), K53XOR(db[(E3)+(32*N)]), K18XOR(db[(E4)+(32*N)]), K45XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K54XOR(db[43*N]), K31XOR(db[44*N]), K32XOR(db[45*N]), K26XOR(db[46*N]), K20XOR(db[47*N]), K11XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K54XOR(db[E0]), K33XOR(db[E1]), K20XOR(db[E2]), K39XOR(db[E3]), K04XOR(db[E4]), K31XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K40XOR(db[11*N]), K48XOR(db[12*N]), K18XOR(db[13*N]), K12XOR(db[14*N]), K06XOR(db[15*N]), K52XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K40XOR(db[(E0)+(32*N)]), K19XOR(db[(E1)+(32*N)]), K06XOR(db[(E2)+(32*N)]), K25XOR(db[(E3)+(32*N)]), K45XOR(db[(E4)+(32*N)]), K48XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K26XOR(db[43*N]), K34XOR(db[44*N]), K04XOR(db[45*N]), K53XOR(db[46*N]), K47XOR(db[47*N]), K38XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K26XOR(db[E0]), K05XOR(db[E1]), K47XOR(db[E2]), K11XOR(db[E3]), K31XOR(db[E4]), K34XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K12XOR(db[11*N]), K20XOR(db[12*N]), K45XOR(db[13*N]), K39XOR(db[14*N]), K33XOR(db[15*N]), K55XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K19XOR(db[(E0)+(32*N)]), K53XOR(db[(E1)+(32*N)]), K40XOR(db[(E2)+(32*N)]), K04XOR(db[(E3)+(32*N)]), K55XOR(db[(E4)+(32*N)]), K27XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K05XOR(db[43*N]), K13XOR(db[44*N]), K38XOR(db[45*N]), K32XOR(db[46*N]), K26XOR(db[47*N]), K48XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			break;
		case 1: 
			s3(K04XOR(db[ 7*N]), K32XOR(db[ 8*N]), K26XOR(db[ 9*N]), K27XOR(db[10*N]), K38XOR(db[11*N]), K54XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		    s2(K34XOR(db[E0]), K55XOR(db[E1]), K05XOR(db[E2]), K13XOR(db[E3]), K18XOR(db[E4]), K40XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K52XOR(db[39*N]), K25XOR(db[40*N]), K19XOR(db[41*N]), K20XOR(db[42*N]), K31XOR(db[43*N]), K47XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		    s2(K27XOR(db[(E0)+(32*N)]), K48XOR(db[(E1)+(32*N)]), K53XOR(db[(E2)+(32*N)]), K06XOR(db[(E3)+(32*N)]), K11XOR(db[(E4)+(32*N)]), K33XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K38XOR(db[ 7*N]), K11XOR(db[ 8*N]), K05XOR(db[ 9*N]), K06XOR(db[10*N]), K48XOR(db[11*N]), K33XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
	        s2(K13XOR(db[E0]), K34XOR(db[E1]), K39XOR(db[E2]), K47XOR(db[E3]), K52XOR(db[E4]), K19XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]);
			__syncthreads();
			s3(K55XOR(db[39*N]), K52XOR(db[40*N]), K46XOR(db[41*N]), K47XOR(db[42*N]), K34XOR(db[43*N]), K19XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
		    s2(K54XOR(db[(E0)+(32*N)]), K20XOR(db[(E1)+(32*N)]), K25XOR(db[(E2)+(32*N)]), K33XOR(db[(E3)+(32*N)]), K38XOR(db[(E4)+(32*N)]), K05XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]);
			__syncthreads();
			s3(K41XOR(db[ 7*N]), K38XOR(db[ 8*N]), K32XOR(db[ 9*N]), K33XOR(db[10*N]), K20XOR(db[11*N]), K05XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
		    s2(K40XOR(db[E0]), K06XOR(db[E1]), K11XOR(db[E2]), K19XOR(db[E3]), K55XOR(db[E4]), K46XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]);
			__syncthreads();
			s3(K27XOR(db[39*N]), K55XOR(db[40*N]), K18XOR(db[41*N]), K19XOR(db[42*N]), K06XOR(db[43*N]), K46XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K26XOR(db[(E0)+(32*N)]), K47XOR(db[(E1)+(32*N)]), K52XOR(db[(E2)+(32*N)]), K05XOR(db[(E3)+(32*N)]), K41XOR(db[(E4)+(32*N)]), K32XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K13XOR(db[ 7*N]), K41XOR(db[ 8*N]), K04XOR(db[ 9*N]), K05XOR(db[10*N]), K47XOR(db[11*N]), K32XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K12XOR(db[E0]), K33XOR(db[E1]), K38XOR(db[E2]), K46XOR(db[E3]), K27XOR(db[E4]), K18XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K54XOR(db[39*N]), K27XOR(db[40*N]), K45XOR(db[41*N]), K46XOR(db[42*N]), K33XOR(db[43*N]), K18XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K53XOR(db[(E0)+(32*N)]), K19XOR(db[(E1)+(32*N)]), K55XOR(db[(E2)+(32*N)]), K32XOR(db[(E3)+(32*N)]), K13XOR(db[(E4)+(32*N)]), K04XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K47XOR(db[ 7*N]), K20XOR(db[ 8*N]), K38XOR(db[ 9*N]), K39XOR(db[10*N]), K26XOR(db[11*N]), K11XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K46XOR(db[E0]), K12XOR(db[E1]), K48XOR(db[E2]), K25XOR(db[E3]), K06XOR(db[E4]), K52XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K33XOR(db[39*N]), K06XOR(db[40*N]), K55XOR(db[41*N]), K25XOR(db[42*N]), K12XOR(db[43*N]), K52XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K32XOR(db[(E0)+(32*N)]), K53XOR(db[(E1)+(32*N)]), K34XOR(db[(E2)+(32*N)]), K11XOR(db[(E3)+(32*N)]), K47XOR(db[(E4)+(32*N)]), K38XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K19XOR(db[ 7*N]), K47XOR(db[ 8*N]), K41XOR(db[ 9*N]), K11XOR(db[10*N]), K53XOR(db[11*N]), K38XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K18XOR(db[E0]), K39XOR(db[E1]), K20XOR(db[E2]), K52XOR(db[E3]), K33XOR(db[E4]), K55XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K05XOR(db[39*N]), K33XOR(db[40*N]), K27XOR(db[41*N]), K52XOR(db[42*N]), K39XOR(db[43*N]), K55XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K04XOR(db[(E0)+(32*N)]), K25XOR(db[(E1)+(32*N)]), K06XOR(db[(E2)+(32*N)]), K38XOR(db[(E3)+(32*N)]), K19XOR(db[(E4)+(32*N)]), K41XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K46XOR(db[ 7*N]), K19XOR(db[ 8*N]), K13XOR(db[ 9*N]), K38XOR(db[10*N]), K25XOR(db[11*N]), K41XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K45XOR(db[E0]), K11XOR(db[E1]), K47XOR(db[E2]), K55XOR(db[E3]), K05XOR(db[E4]), K27XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K32XOR(db[39*N]), K05XOR(db[40*N]), K54XOR(db[41*N]), K55XOR(db[42*N]), K11XOR(db[43*N]), K27XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K31XOR(db[(E0)+(32*N)]), K52XOR(db[(E1)+(32*N)]), K33XOR(db[(E2)+(32*N)]), K41XOR(db[(E3)+(32*N)]), K46XOR(db[(E4)+(32*N)]), K13XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K18XOR(db[ 7*N]), K46XOR(db[ 8*N]), K40XOR(db[ 9*N]), K41XOR(db[10*N]), K52XOR(db[11*N]), K13XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K48XOR(db[E0]), K38XOR(db[E1]), K19XOR(db[E2]), K27XOR(db[E3]), K32XOR(db[E4]), K54XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K11XOR(db[39*N]), K39XOR(db[40*N]), K33XOR(db[41*N]), K34XOR(db[42*N]), K45XOR(db[43*N]), K06XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K41XOR(db[(E0)+(32*N)]), K31XOR(db[(E1)+(32*N)]), K12XOR(db[(E2)+(32*N)]), K20XOR(db[(E3)+(32*N)]), K25XOR(db[(E4)+(32*N)]), K47XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			break;
		case 2: 
			s5(K15XOR(db[E0]), K24XOR(db[E1]), K28XOR(db[E2]), K43XOR(db[E3]), K30XOR(db[E4]), K03XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K37XOR(db[27*N]), K08XOR(db[28*N]), K09XOR(db[29*N]), K50XOR(db[30*N]), K42XOR(db[31*N]), K21XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]);
			__syncthreads();
			s5(K08XOR(db[(E0)+(32*N)]), K17XOR(db[(E1)+(32*N)]), K21XOR(db[(E2)+(32*N)]), K36XOR(db[(E3)+(32*N)]), K23XOR(db[(E4)+(32*N)]), K49XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K30XOR(db[59*N]), K01XOR(db[60*N]), K02XOR(db[61*N]), K43XOR(db[62*N]), K35XOR(db[63*N]), K14XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]);
			__syncthreads();
			s5(K51XOR(db[E0]), K03XOR(db[E1]), K07XOR(db[E2]), K22XOR(db[E3]), K09XOR(db[E4]), K35XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
	        s8(K16XOR(db[27*N]), K44XOR(db[28*N]), K17XOR(db[29*N]), K29XOR(db[30*N]), K21XOR(db[31*N]), K00XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]);
			__syncthreads();
			s5(K37XOR(db[(E0)+(32*N)]), K42XOR(db[(E1)+(32*N)]), K50XOR(db[(E2)+(32*N)]), K08XOR(db[(E3)+(32*N)]), K24XOR(db[(E4)+(32*N)]), K21XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
		    s8(K02XOR(db[59*N]), K30XOR(db[60*N]), K03XOR(db[61*N]), K15XOR(db[62*N]), K07XOR(db[63*N]), K43XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]);
			__syncthreads();
			s5(K23XOR(db[E0]), K28XOR(db[E1]), K36XOR(db[E2]), K51XOR(db[E3]), K10XOR(db[E4]), K07XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
		    s8(K17XOR(db[27*N]), K16XOR(db[28*N]), K42XOR(db[29*N]), K01XOR(db[30*N]), K50XOR(db[31*N]), K29XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]);
			__syncthreads();
			s5(K09XOR(db[(E0)+(32*N)]), K14XOR(db[(E1)+(32*N)]), K22XOR(db[(E2)+(32*N)]), K37XOR(db[(E3)+(32*N)]), K49XOR(db[(E4)+(32*N)]), K50XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K03XOR(db[59*N]), K02XOR(db[60*N]), K28XOR(db[61*N]), K44XOR(db[62*N]), K36XOR(db[63*N]), K15XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K24XOR(db[E0]), K00XOR(db[E1]), K08XOR(db[E2]), K23XOR(db[E3]), K35XOR(db[E4]), K36XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K42XOR(db[27*N]), K17XOR(db[28*N]), K14XOR(db[29*N]), K30XOR(db[30*N]), K22XOR(db[31*N]), K01XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K10XOR(db[(E0)+(32*N)]), K43XOR(db[(E1)+(32*N)]), K51XOR(db[(E2)+(32*N)]), K09XOR(db[(E3)+(32*N)]), K21XOR(db[(E4)+(32*N)]), K22XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K28XOR(db[59*N]), K03XOR(db[60*N]), K00XOR(db[61*N]), K16XOR(db[62*N]), K08XOR(db[63*N]), K44XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K03XOR(db[E0]), K36XOR(db[E1]), K44XOR(db[E2]), K02XOR(db[E3]), K14XOR(db[E4]), K15XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K21XOR(db[27*N]), K49XOR(db[28*N]), K50XOR(db[29*N]), K09XOR(db[30*N]), K01XOR(db[31*N]), K37XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K42XOR(db[(E0)+(32*N)]), K22XOR(db[(E1)+(32*N)]), K30XOR(db[(E2)+(32*N)]), K17XOR(db[(E3)+(32*N)]), K00XOR(db[(E4)+(32*N)]), K01XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K07XOR(db[59*N]), K35XOR(db[60*N]), K36XOR(db[61*N]), K24XOR(db[62*N]), K44XOR(db[63*N]), K23XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K28XOR(db[E0]), K08XOR(db[E1]), K16XOR(db[E2]), K03XOR(db[E3]), K43XOR(db[E4]), K44XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K50XOR(db[27*N]), K21XOR(db[28*N]), K22XOR(db[29*N]), K10XOR(db[30*N]), K30XOR(db[31*N]), K09XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K14XOR(db[(E0)+(32*N)]), K51XOR(db[(E1)+(32*N)]), K02XOR(db[(E2)+(32*N)]), K42XOR(db[(E3)+(32*N)]), K29XOR(db[(E4)+(32*N)]), K30XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K36XOR(db[59*N]), K07XOR(db[60*N]), K08XOR(db[61*N]), K49XOR(db[62*N]), K16XOR(db[63*N]), K24XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K00XOR(db[E0]), K37XOR(db[E1]), K17XOR(db[E2]), K28XOR(db[E3]), K15XOR(db[E4]), K16XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K22XOR(db[27*N]), K50XOR(db[28*N]), K51XOR(db[29*N]), K35XOR(db[30*N]), K02XOR(db[31*N]), K10XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K43XOR(db[(E0)+(32*N)]), K23XOR(db[(E1)+(32*N)]), K03XOR(db[(E2)+(32*N)]), K14XOR(db[(E3)+(32*N)]), K01XOR(db[(E4)+(32*N)]), K02XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K08XOR(db[59*N]), K36XOR(db[60*N]), K37XOR(db[61*N]), K21XOR(db[62*N]), K17XOR(db[63*N]), K49XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K29XOR(db[E0]), K09XOR(db[E1]), K42XOR(db[E2]), K00XOR(db[E3]), K44XOR(db[E4]), K17XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K51XOR(db[27*N]), K22XOR(db[28*N]), K23XOR(db[29*N]), K07XOR(db[30*N]), K03XOR(db[31*N]), K35XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K22XOR(db[(E0)+(32*N)]), K02XOR(db[(E1)+(32*N)]), K35XOR(db[(E2)+(32*N)]), K50XOR(db[(E3)+(32*N)]), K37XOR(db[(E4)+(32*N)]), K10XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K44XOR(db[59*N]), K15XOR(db[60*N]), K16XOR(db[61*N]), K00XOR(db[62*N]), K49XOR(db[63*N]), K28XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			break;
		case 3: 
			s7(K51XOR(db[23*N]), K16XOR(db[24*N]), K29XOR(db[25*N]), K49XOR(db[26*N]), K07XOR(db[27*N]), K17XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K35XOR(db[E0]), K22XOR(db[E1]), K02XOR(db[E2]), K44XOR(db[E3]), K14XOR(db[E4]), K23XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K44XOR(db[55*N]), K09XOR(db[56*N]), K22XOR(db[57*N]), K42XOR(db[58*N]), K00XOR(db[59*N]), K10XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K28XOR(db[(E0)+(32*N)]), K15XOR(db[(E1)+(32*N)]), K24XOR(db[(E2)+(32*N)]), K37XOR(db[(E3)+(32*N)]), K07XOR(db[(E4)+(32*N)]), K16XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]);
			__syncthreads();
			s7(K30XOR(db[23*N]), K24XOR(db[24*N]), K08XOR(db[25*N]), K28XOR(db[26*N]), K43XOR(db[27*N]), K49XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
	        s6(K14XOR(db[E0]), K01XOR(db[E1]), K10XOR(db[E2]), K23XOR(db[E3]), K50XOR(db[E4]), K02XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]);
			__syncthreads();
			s7(K16XOR(db[55*N]), K10XOR(db[56*N]), K51XOR(db[57*N]), K14XOR(db[58*N]), K29XOR(db[59*N]), K35XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K00XOR(db[(E0)+(32*N)]), K44XOR(db[(E1)+(32*N)]), K49XOR(db[(E2)+(32*N)]), K09XOR(db[(E3)+(32*N)]), K36XOR(db[(E4)+(32*N)]), K17XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]);
			__syncthreads();
			s7(K02XOR(db[23*N]), K49XOR(db[24*N]), K37XOR(db[25*N]), K00XOR(db[26*N]), K15XOR(db[27*N]), K21XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
		    s6(K43XOR(db[E0]), K30XOR(db[E1]), K35XOR(db[E2]), K24XOR(db[E3]), K22XOR(db[E4]), K03XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]);
			__syncthreads();
			s7(K17XOR(db[55*N]), K35XOR(db[56*N]), K23XOR(db[57*N]), K43XOR(db[58*N]), K01XOR(db[59*N]), K07XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K29XOR(db[(E0)+(32*N)]), K16XOR(db[(E1)+(32*N)]), K21XOR(db[(E2)+(32*N)]), K10XOR(db[(E3)+(32*N)]), K08XOR(db[(E4)+(32*N)]), K42XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K03XOR(db[23*N]), K21XOR(db[24*N]), K09XOR(db[25*N]), K29XOR(db[26*N]), K44XOR(db[27*N]), K50XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K15XOR(db[E0]), K02XOR(db[E1]), K07XOR(db[E2]), K49XOR(db[E3]), K51XOR(db[E4]), K28XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K42XOR(db[55*N]), K07XOR(db[56*N]), K24XOR(db[57*N]), K15XOR(db[58*N]), K30XOR(db[59*N]), K36XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K01XOR(db[(E0)+(32*N)]), K17XOR(db[(E1)+(32*N)]), K50XOR(db[(E2)+(32*N)]), K35XOR(db[(E3)+(32*N)]), K37XOR(db[(E4)+(32*N)]), K14XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K35XOR(db[23*N]), K00XOR(db[24*N]), K17XOR(db[25*N]), K08XOR(db[26*N]), K23XOR(db[27*N]), K29XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K51XOR(db[E0]), K10XOR(db[E1]), K43XOR(db[E2]), K28XOR(db[E3]), K30XOR(db[E4]), K07XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K21XOR(db[55*N]), K43XOR(db[56*N]), K03XOR(db[57*N]), K51XOR(db[58*N]), K09XOR(db[59*N]), K15XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K37XOR(db[(E0)+(32*N)]), K49XOR(db[(E1)+(32*N)]), K29XOR(db[(E2)+(32*N)]), K14XOR(db[(E3)+(32*N)]), K16XOR(db[(E4)+(32*N)]), K50XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K07XOR(db[23*N]), K29XOR(db[24*N]), K42XOR(db[25*N]), K37XOR(db[26*N]), K24XOR(db[27*N]), K01XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K23XOR(db[E0]), K35XOR(db[E1]), K15XOR(db[E2]), K00XOR(db[E3]), K02XOR(db[E4]), K36XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K50XOR(db[55*N]), K15XOR(db[56*N]), K28XOR(db[57*N]), K23XOR(db[58*N]), K10XOR(db[59*N]), K44XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K09XOR(db[(E0)+(32*N)]), K21XOR(db[(E1)+(32*N)]), K01XOR(db[(E2)+(32*N)]), K43XOR(db[(E3)+(32*N)]), K17XOR(db[(E4)+(32*N)]), K22XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K36XOR(db[23*N]), K01XOR(db[24*N]), K14XOR(db[25*N]), K09XOR(db[26*N]), K49XOR(db[27*N]), K30XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K24XOR(db[E0]), K07XOR(db[E1]), K44XOR(db[E2]), K29XOR(db[E3]), K03XOR(db[E4]), K08XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K22XOR(db[55*N]), K44XOR(db[56*N]), K00XOR(db[57*N]), K24XOR(db[58*N]), K35XOR(db[59*N]), K16XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K10XOR(db[(E0)+(32*N)]), K50XOR(db[(E1)+(32*N)]), K30XOR(db[(E2)+(32*N)]), K15XOR(db[(E3)+(32*N)]), K42XOR(db[(E4)+(32*N)]), K51XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K08XOR(db[23*N]), K30XOR(db[24*N]), K43XOR(db[25*N]), K10XOR(db[26*N]), K21XOR(db[27*N]), K02XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K49XOR(db[E0]), K36XOR(db[E1]), K16XOR(db[E2]), K01XOR(db[E3]), K28XOR(db[E4]), K37XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K01XOR(db[55*N]), K23XOR(db[56*N]), K36XOR(db[57*N]), K03XOR(db[58*N]), K14XOR(db[59*N]), K24XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K42XOR(db[(E0)+(32*N)]), K29XOR(db[(E1)+(32*N)]), K09XOR(db[(E2)+(32*N)]), K51XOR(db[(E3)+(32*N)]), K21XOR(db[(E4)+(32*N)]), K30XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			break;
		}
		__syncthreads();

		if (i >= 12)
			break;

		// ROUND_B(-48);
		switch (threadIdx.y) {
		case 0:
			s1(K12XOR(db[(E0)+(32*N)]), K46XOR(db[(E1)+(32*N)]), K33XOR(db[(E2)+(32*N)]), K52XOR(db[(E3)+(32*N)]), K48XOR(db[(E4)+(32*N)]), K20XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K53XOR(db[43*N]), K06XOR(db[44*N]), K31XOR(db[45*N]), K25XOR(db[46*N]), K19XOR(db[47*N]), K41XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K05XOR(db[E0]), K39XOR(db[E1]), K26XOR(db[E2]), K45XOR(db[E3]), K41XOR(db[E4]), K13XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K46XOR(db[11*N]), K54XOR(db[12*N]), K55XOR(db[13*N]), K18XOR(db[14*N]), K12XOR(db[15*N]), K34XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K46XOR(db[(E0)+(32*N)]), K25XOR(db[(E1)+(32*N)]), K12XOR(db[(E2)+(32*N)]), K31XOR(db[(E3)+(32*N)]), K27XOR(db[(E4)+(32*N)]), K54XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K32XOR(db[43*N]), K40XOR(db[44*N]), K41XOR(db[45*N]), K04XOR(db[46*N]), K53XOR(db[47*N]), K20XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K32XOR(db[E0]), K11XOR(db[E1]), K53XOR(db[E2]), K48XOR(db[E3]), K13XOR(db[E4]), K40XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K18XOR(db[11*N]), K26XOR(db[12*N]), K27XOR(db[13*N]), K45XOR(db[14*N]), K39XOR(db[15*N]), K06XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K18XOR(db[(E0)+(32*N)]), K52XOR(db[(E1)+(32*N)]), K39XOR(db[(E2)+(32*N)]), K34XOR(db[(E3)+(32*N)]), K54XOR(db[(E4)+(32*N)]), K26XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K04XOR(db[43*N]), K12XOR(db[44*N]), K13XOR(db[45*N]), K31XOR(db[46*N]), K25XOR(db[47*N]), K47XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K04XOR(db[E0]), K38XOR(db[E1]), K25XOR(db[E2]), K20XOR(db[E3]), K40XOR(db[E4]), K12XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K45XOR(db[11*N]), K53XOR(db[12*N]), K54XOR(db[13*N]), K48XOR(db[14*N]), K11XOR(db[15*N]), K33XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K45XOR(db[(E0)+(32*N)]), K55XOR(db[(E1)+(32*N)]), K11XOR(db[(E2)+(32*N)]), K06XOR(db[(E3)+(32*N)]), K26XOR(db[(E4)+(32*N)]), K53XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K31XOR(db[43*N]), K39XOR(db[44*N]), K40XOR(db[45*N]), K34XOR(db[46*N]), K52XOR(db[47*N]), K19XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K31XOR(db[E0]), K41XOR(db[E1]), K52XOR(db[E2]), K47XOR(db[E3]), K12XOR(db[E4]), K39XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K48XOR(db[11*N]), K25XOR(db[12*N]), K26XOR(db[13*N]), K20XOR(db[14*N]), K38XOR(db[15*N]), K05XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K55XOR(db[(E0)+(32*N)]), K34XOR(db[(E1)+(32*N)]), K45XOR(db[(E2)+(32*N)]), K40XOR(db[(E3)+(32*N)]), K05XOR(db[(E4)+(32*N)]), K32XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K41XOR(db[43*N]), K18XOR(db[44*N]), K19XOR(db[45*N]), K13XOR(db[46*N]), K31XOR(db[47*N]), K53XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K41XOR(db[E0]), K20XOR(db[E1]), K31XOR(db[E2]), K26XOR(db[E3]), K46XOR(db[E4]), K18XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K27XOR(db[11*N]), K04XOR(db[12*N]), K05XOR(db[13*N]), K54XOR(db[14*N]), K48XOR(db[15*N]), K39XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K27XOR(db[(E0)+(32*N)]), K06XOR(db[(E1)+(32*N)]), K48XOR(db[(E2)+(32*N)]), K12XOR(db[(E3)+(32*N)]), K32XOR(db[(E4)+(32*N)]), K04XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K13XOR(db[43*N]), K45XOR(db[44*N]), K46XOR(db[45*N]), K40XOR(db[46*N]), K34XOR(db[47*N]), K25XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K13XOR(db[E0]), K47XOR(db[E1]), K34XOR(db[E2]), K53XOR(db[E3]), K18XOR(db[E4]), K45XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K54XOR(db[11*N]), K31XOR(db[12*N]), K32XOR(db[13*N]), K26XOR(db[14*N]), K20XOR(db[15*N]), K11XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K54XOR(db[(E0)+(32*N)]), K33XOR(db[(E1)+(32*N)]), K20XOR(db[(E2)+(32*N)]), K39XOR(db[(E3)+(32*N)]), K04XOR(db[(E4)+(32*N)]), K31XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K40XOR(db[43*N]), K48XOR(db[44*N]), K18XOR(db[45*N]), K12XOR(db[46*N]), K06XOR(db[47*N]), K52XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K40XOR(db[E0]), K19XOR(db[E1]), K06XOR(db[E2]), K25XOR(db[E3]), K45XOR(db[E4]), K48XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K26XOR(db[11*N]), K34XOR(db[12*N]), K04XOR(db[13*N]), K53XOR(db[14*N]), K47XOR(db[15*N]), K38XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			__syncthreads();
			s1(K26XOR(db[(E0)+(32*N)]), K05XOR(db[(E1)+(32*N)]), K47XOR(db[(E2)+(32*N)]), K11XOR(db[(E3)+(32*N)]), K31XOR(db[(E4)+(32*N)]), K34XOR(db[(E5)+(32*N)]), &db[ 8*N], &db[16*N], &db[22*N], &db[30*N]);
			s4(K12XOR(db[43*N]), K20XOR(db[44*N]), K45XOR(db[45*N]), K39XOR(db[46*N]), K33XOR(db[47*N]), K55XOR(db[48*N]), &db[25*N], &db[19*N], &db[ 9*N], &db[ 0*N]); 
			__syncthreads();
			s1(K19XOR(db[E0]), K53XOR(db[E1]), K40XOR(db[E2]), K04XOR(db[E3]), K55XOR(db[E4]), K27XOR(db[E5]), &db[40*N], &db[48*N], &db[54*N], &db[62*N]);
			s4(K05XOR(db[11*N]), K13XOR(db[12*N]), K38XOR(db[13*N]), K32XOR(db[14*N]), K26XOR(db[15*N]), K48XOR(db[16*N]), &db[57*N], &db[51*N], &db[41*N], &db[32*N]); 
			break;
		case 1:
			s3(K04XOR(db[39*N]), K32XOR(db[40*N]), K26XOR(db[41*N]), K27XOR(db[42*N]), K38XOR(db[43*N]), K54XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K34XOR(db[(E0)+(32*N)]), K55XOR(db[(E1)+(32*N)]), K05XOR(db[(E2)+(32*N)]), K13XOR(db[(E3)+(32*N)]), K18XOR(db[(E4)+(32*N)]), K40XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K52XOR(db[ 7*N]), K25XOR(db[ 8*N]), K19XOR(db[ 9*N]), K20XOR(db[10*N]), K31XOR(db[11*N]), K47XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K27XOR(db[E0]), K48XOR(db[E1]), K53XOR(db[E2]), K06XOR(db[E3]), K11XOR(db[E4]), K33XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K38XOR(db[39*N]), K11XOR(db[40*N]), K05XOR(db[41*N]), K06XOR(db[42*N]), K48XOR(db[43*N]), K33XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K13XOR(db[(E0)+(32*N)]), K34XOR(db[(E1)+(32*N)]), K39XOR(db[(E2)+(32*N)]), K47XOR(db[(E3)+(32*N)]), K52XOR(db[(E4)+(32*N)]), K19XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K55XOR(db[ 7*N]), K52XOR(db[ 8*N]), K46XOR(db[ 9*N]), K47XOR(db[10*N]), K34XOR(db[11*N]), K19XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K54XOR(db[E0]), K20XOR(db[E1]), K25XOR(db[E2]), K33XOR(db[E3]), K38XOR(db[E4]), K05XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K41XOR(db[39*N]), K38XOR(db[40*N]), K32XOR(db[41*N]), K33XOR(db[42*N]), K20XOR(db[43*N]), K05XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K40XOR(db[(E0)+(32*N)]), K06XOR(db[(E1)+(32*N)]), K11XOR(db[(E2)+(32*N)]), K19XOR(db[(E3)+(32*N)]), K55XOR(db[(E4)+(32*N)]), K46XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K27XOR(db[ 7*N]), K55XOR(db[ 8*N]), K18XOR(db[ 9*N]), K19XOR(db[10*N]), K06XOR(db[11*N]), K46XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K26XOR(db[E0]), K47XOR(db[E1]), K52XOR(db[E2]), K05XOR(db[E3]), K41XOR(db[E4]), K32XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K13XOR(db[39*N]), K41XOR(db[40*N]), K04XOR(db[41*N]), K05XOR(db[42*N]), K47XOR(db[43*N]), K32XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K12XOR(db[(E0)+(32*N)]), K33XOR(db[(E1)+(32*N)]), K38XOR(db[(E2)+(32*N)]), K46XOR(db[(E3)+(32*N)]), K27XOR(db[(E4)+(32*N)]), K18XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K54XOR(db[ 7*N]), K27XOR(db[ 8*N]), K45XOR(db[ 9*N]), K46XOR(db[10*N]), K33XOR(db[11*N]), K18XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K53XOR(db[E0]), K19XOR(db[E1]), K55XOR(db[E2]), K32XOR(db[E3]), K13XOR(db[E4]), K04XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K47XOR(db[39*N]), K20XOR(db[40*N]), K38XOR(db[41*N]), K39XOR(db[42*N]), K26XOR(db[43*N]), K11XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K46XOR(db[(E0)+(32*N)]), K12XOR(db[(E1)+(32*N)]), K48XOR(db[(E2)+(32*N)]), K25XOR(db[(E3)+(32*N)]), K06XOR(db[(E4)+(32*N)]), K52XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K33XOR(db[ 7*N]), K06XOR(db[ 8*N]), K55XOR(db[ 9*N]), K25XOR(db[10*N]), K12XOR(db[11*N]), K52XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K32XOR(db[E0]), K53XOR(db[E1]), K34XOR(db[E2]), K11XOR(db[E3]), K47XOR(db[E4]), K38XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K19XOR(db[39*N]), K47XOR(db[40*N]), K41XOR(db[41*N]), K11XOR(db[42*N]), K53XOR(db[43*N]), K38XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K18XOR(db[(E0)+(32*N)]), K39XOR(db[(E1)+(32*N)]), K20XOR(db[(E2)+(32*N)]), K52XOR(db[(E3)+(32*N)]), K33XOR(db[(E4)+(32*N)]), K55XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K05XOR(db[ 7*N]), K33XOR(db[ 8*N]), K27XOR(db[ 9*N]), K52XOR(db[10*N]), K39XOR(db[11*N]), K55XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K04XOR(db[E0]), K25XOR(db[E1]), K06XOR(db[E2]), K38XOR(db[E3]), K19XOR(db[E4]), K41XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K46XOR(db[39*N]), K19XOR(db[40*N]), K13XOR(db[41*N]), K38XOR(db[42*N]), K25XOR(db[43*N]), K41XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K45XOR(db[(E0)+(32*N)]), K11XOR(db[(E1)+(32*N)]), K47XOR(db[(E2)+(32*N)]), K55XOR(db[(E3)+(32*N)]), K05XOR(db[(E4)+(32*N)]), K27XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K32XOR(db[ 7*N]), K05XOR(db[ 8*N]), K54XOR(db[ 9*N]), K55XOR(db[10*N]), K11XOR(db[11*N]), K27XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K31XOR(db[E0]), K52XOR(db[E1]), K33XOR(db[E2]), K41XOR(db[E3]), K46XOR(db[E4]), K13XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			__syncthreads();
			s3(K18XOR(db[39*N]), K46XOR(db[40*N]), K40XOR(db[41*N]), K41XOR(db[42*N]), K52XOR(db[43*N]), K13XOR(db[44*N]), &db[23*N], &db[15*N], &db[29*N], &db[ 5*N]);
			s2(K48XOR(db[(E0)+(32*N)]), K38XOR(db[(E1)+(32*N)]), K19XOR(db[(E2)+(32*N)]), K27XOR(db[(E3)+(32*N)]), K32XOR(db[(E4)+(32*N)]), K54XOR(db[(E5)+(32*N)]), &db[12*N], &db[27*N], &db[ 1*N], &db[17*N]); 
			__syncthreads();
			s3(K11XOR(db[ 7*N]), K39XOR(db[ 8*N]), K33XOR(db[ 9*N]), K34XOR(db[10*N]), K45XOR(db[11*N]), K06XOR(db[12*N]), &db[55*N], &db[47*N], &db[61*N], &db[37*N]);
			s2(K41XOR(db[E0]), K31XOR(db[E1]), K12XOR(db[E2]), K20XOR(db[E3]), K25XOR(db[E4]), K47XOR(db[E5]), &db[44*N], &db[59*N], &db[33*N], &db[49*N]); 
			break;
		case 2:
			s5(K15XOR(db[(E0)+(32*N)]), K24XOR(db[(E1)+(32*N)]), K28XOR(db[(E2)+(32*N)]), K43XOR(db[(E3)+(32*N)]), K30XOR(db[(E4)+(32*N)]), K03XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K37XOR(db[59*N]), K08XOR(db[60*N]), K09XOR(db[61*N]), K50XOR(db[62*N]), K42XOR(db[63*N]), K21XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K08XOR(db[E0]), K17XOR(db[E1]), K21XOR(db[E2]), K36XOR(db[E3]), K23XOR(db[E4]), K49XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K30XOR(db[27*N]), K01XOR(db[28*N]), K02XOR(db[29*N]), K43XOR(db[30*N]), K35XOR(db[31*N]), K14XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K51XOR(db[(E0)+(32*N)]), K03XOR(db[(E1)+(32*N)]), K07XOR(db[(E2)+(32*N)]), K22XOR(db[(E3)+(32*N)]), K09XOR(db[(E4)+(32*N)]), K35XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K16XOR(db[59*N]), K44XOR(db[60*N]), K17XOR(db[61*N]), K29XOR(db[62*N]), K21XOR(db[63*N]), K00XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K37XOR(db[E0]), K42XOR(db[E1]), K50XOR(db[E2]), K08XOR(db[E3]), K24XOR(db[E4]), K21XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K02XOR(db[27*N]), K30XOR(db[28*N]), K03XOR(db[29*N]), K15XOR(db[30*N]), K07XOR(db[31*N]), K43XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K23XOR(db[(E0)+(32*N)]), K28XOR(db[(E1)+(32*N)]), K36XOR(db[(E2)+(32*N)]), K51XOR(db[(E3)+(32*N)]), K10XOR(db[(E4)+(32*N)]), K07XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K17XOR(db[59*N]), K16XOR(db[60*N]), K42XOR(db[61*N]), K01XOR(db[62*N]), K50XOR(db[63*N]), K29XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K09XOR(db[E0]), K14XOR(db[E1]), K22XOR(db[E2]), K37XOR(db[E3]), K49XOR(db[E4]), K50XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K03XOR(db[27*N]), K02XOR(db[28*N]), K28XOR(db[29*N]), K44XOR(db[30*N]), K36XOR(db[31*N]), K15XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K24XOR(db[(E0)+(32*N)]), K00XOR(db[(E1)+(32*N)]), K08XOR(db[(E2)+(32*N)]), K23XOR(db[(E3)+(32*N)]), K35XOR(db[(E4)+(32*N)]), K36XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K42XOR(db[59*N]), K17XOR(db[60*N]), K14XOR(db[61*N]), K30XOR(db[62*N]), K22XOR(db[63*N]), K01XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K10XOR(db[E0]), K43XOR(db[E1]), K51XOR(db[E2]), K09XOR(db[E3]), K21XOR(db[E4]), K22XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K28XOR(db[27*N]), K03XOR(db[28*N]), K00XOR(db[29*N]), K16XOR(db[30*N]), K08XOR(db[31*N]), K44XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K03XOR(db[(E0)+(32*N)]), K36XOR(db[(E1)+(32*N)]), K44XOR(db[(E2)+(32*N)]), K02XOR(db[(E3)+(32*N)]), K14XOR(db[(E4)+(32*N)]), K15XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K21XOR(db[59*N]), K49XOR(db[60*N]), K50XOR(db[61*N]), K09XOR(db[62*N]), K01XOR(db[63*N]), K37XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K42XOR(db[E0]), K22XOR(db[E1]), K30XOR(db[E2]), K17XOR(db[E3]), K00XOR(db[E4]), K01XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K07XOR(db[27*N]), K35XOR(db[28*N]), K36XOR(db[29*N]), K24XOR(db[30*N]), K44XOR(db[31*N]), K23XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K28XOR(db[(E0)+(32*N)]), K08XOR(db[(E1)+(32*N)]), K16XOR(db[(E2)+(32*N)]), K03XOR(db[(E3)+(32*N)]), K43XOR(db[(E4)+(32*N)]), K44XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K50XOR(db[59*N]), K21XOR(db[60*N]), K22XOR(db[61*N]), K10XOR(db[62*N]), K30XOR(db[63*N]), K09XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K14XOR(db[E0]), K51XOR(db[E1]), K02XOR(db[E2]), K42XOR(db[E3]), K29XOR(db[E4]), K30XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K36XOR(db[27*N]), K07XOR(db[28*N]), K08XOR(db[29*N]), K49XOR(db[30*N]), K16XOR(db[31*N]), K24XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K00XOR(db[(E0)+(32*N)]), K37XOR(db[(E1)+(32*N)]), K17XOR(db[(E2)+(32*N)]), K28XOR(db[(E3)+(32*N)]), K15XOR(db[(E4)+(32*N)]), K16XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K22XOR(db[59*N]), K50XOR(db[60*N]), K51XOR(db[61*N]), K35XOR(db[62*N]), K02XOR(db[63*N]), K10XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K43XOR(db[E0]), K23XOR(db[E1]), K03XOR(db[E2]), K14XOR(db[E3]), K01XOR(db[E4]), K02XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K08XOR(db[27*N]), K36XOR(db[28*N]), K37XOR(db[29*N]), K21XOR(db[30*N]), K17XOR(db[31*N]), K49XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			__syncthreads();
			s5(K29XOR(db[(E0)+(32*N)]), K09XOR(db[(E1)+(32*N)]), K42XOR(db[(E2)+(32*N)]), K00XOR(db[(E3)+(32*N)]), K44XOR(db[(E4)+(32*N)]), K17XOR(db[(E5)+(32*N)]), &db[ 7*N], &db[13*N], &db[24*N], &db[ 2*N]);
			s8(K51XOR(db[59*N]), K22XOR(db[60*N]), K23XOR(db[61*N]), K07XOR(db[62*N]), K03XOR(db[63*N]), K35XOR(db[32*N]), &db[ 4*N], &db[26*N], &db[14*N], &db[20*N]); 
			__syncthreads();
			s5(K22XOR(db[E0]), K02XOR(db[E1]), K35XOR(db[E2]), K50XOR(db[E3]), K37XOR(db[E4]), K10XOR(db[E5]), &db[39*N], &db[45*N], &db[56*N], &db[34*N]);
			s8(K44XOR(db[27*N]), K15XOR(db[28*N]), K16XOR(db[29*N]), K00XOR(db[30*N]), K49XOR(db[31*N]), K28XOR(db[ 0*N]), &db[36*N], &db[58*N], &db[46*N], &db[52*N]); 
			break;
		case 3: 
			s7(K51XOR(db[55*N]), K16XOR(db[56*N]), K29XOR(db[57*N]), K49XOR(db[58*N]), K07XOR(db[59*N]), K17XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K35XOR(db[(E0)+(32*N)]), K22XOR(db[(E1)+(32*N)]), K02XOR(db[(E2)+(32*N)]), K44XOR(db[(E3)+(32*N)]), K14XOR(db[(E4)+(32*N)]), K23XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K44XOR(db[23*N]), K09XOR(db[24*N]), K22XOR(db[25*N]), K42XOR(db[26*N]), K00XOR(db[27*N]), K10XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K28XOR(db[E0]), K15XOR(db[E1]), K24XOR(db[E2]), K37XOR(db[E3]), K07XOR(db[E4]), K16XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K30XOR(db[55*N]), K24XOR(db[56*N]), K08XOR(db[57*N]), K28XOR(db[58*N]), K43XOR(db[59*N]), K49XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K14XOR(db[(E0)+(32*N)]), K01XOR(db[(E1)+(32*N)]), K10XOR(db[(E2)+(32*N)]), K23XOR(db[(E3)+(32*N)]), K50XOR(db[(E4)+(32*N)]), K02XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K16XOR(db[23*N]), K10XOR(db[24*N]), K51XOR(db[25*N]), K14XOR(db[26*N]), K29XOR(db[27*N]), K35XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K00XOR(db[E0]), K44XOR(db[E1]), K49XOR(db[E2]), K09XOR(db[E3]), K36XOR(db[E4]), K17XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K02XOR(db[55*N]), K49XOR(db[56*N]), K37XOR(db[57*N]), K00XOR(db[58*N]), K15XOR(db[59*N]), K21XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K43XOR(db[(E0)+(32*N)]), K30XOR(db[(E1)+(32*N)]), K35XOR(db[(E2)+(32*N)]), K24XOR(db[(E3)+(32*N)]), K22XOR(db[(E4)+(32*N)]), K03XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K17XOR(db[23*N]), K35XOR(db[24*N]), K23XOR(db[25*N]), K43XOR(db[26*N]), K01XOR(db[27*N]), K07XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K29XOR(db[E0]), K16XOR(db[E1]), K21XOR(db[E2]), K10XOR(db[E3]), K08XOR(db[E4]), K42XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K03XOR(db[55*N]), K21XOR(db[56*N]), K09XOR(db[57*N]), K29XOR(db[58*N]), K44XOR(db[59*N]), K50XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K15XOR(db[(E0)+(32*N)]), K02XOR(db[(E1)+(32*N)]), K07XOR(db[(E2)+(32*N)]), K49XOR(db[(E3)+(32*N)]), K51XOR(db[(E4)+(32*N)]), K28XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K42XOR(db[23*N]), K07XOR(db[24*N]), K24XOR(db[25*N]), K15XOR(db[26*N]), K30XOR(db[27*N]), K36XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K01XOR(db[E0]), K17XOR(db[E1]), K50XOR(db[E2]), K35XOR(db[E3]), K37XOR(db[E4]), K14XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K35XOR(db[55*N]), K00XOR(db[56*N]), K17XOR(db[57*N]), K08XOR(db[58*N]), K23XOR(db[59*N]), K29XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K51XOR(db[(E0)+(32*N)]), K10XOR(db[(E1)+(32*N)]), K43XOR(db[(E2)+(32*N)]), K28XOR(db[(E3)+(32*N)]), K30XOR(db[(E4)+(32*N)]), K07XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K21XOR(db[23*N]), K43XOR(db[24*N]), K03XOR(db[25*N]), K51XOR(db[26*N]), K09XOR(db[27*N]), K15XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K37XOR(db[E0]), K49XOR(db[E1]), K29XOR(db[E2]), K14XOR(db[E3]), K16XOR(db[E4]), K50XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K07XOR(db[55*N]), K29XOR(db[56*N]), K42XOR(db[57*N]), K37XOR(db[58*N]), K24XOR(db[59*N]), K01XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K23XOR(db[(E0)+(32*N)]), K35XOR(db[(E1)+(32*N)]), K15XOR(db[(E2)+(32*N)]), K00XOR(db[(E3)+(32*N)]), K02XOR(db[(E4)+(32*N)]), K36XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K50XOR(db[23*N]), K15XOR(db[24*N]), K28XOR(db[25*N]), K23XOR(db[26*N]), K10XOR(db[27*N]), K44XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K09XOR(db[E0]), K21XOR(db[E1]), K01XOR(db[E2]), K43XOR(db[E3]), K17XOR(db[E4]), K22XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K36XOR(db[55*N]), K01XOR(db[56*N]), K14XOR(db[57*N]), K09XOR(db[58*N]), K49XOR(db[59*N]), K30XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K24XOR(db[(E0)+(32*N)]), K07XOR(db[(E1)+(32*N)]), K44XOR(db[(E2)+(32*N)]), K29XOR(db[(E3)+(32*N)]), K03XOR(db[(E4)+(32*N)]), K08XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K22XOR(db[23*N]), K44XOR(db[24*N]), K00XOR(db[25*N]), K24XOR(db[26*N]), K35XOR(db[27*N]), K16XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K10XOR(db[E0]), K50XOR(db[E1]), K30XOR(db[E2]), K15XOR(db[E3]), K42XOR(db[E4]), K51XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			__syncthreads();
			s7(K08XOR(db[55*N]), K30XOR(db[56*N]), K43XOR(db[57*N]), K10XOR(db[58*N]), K21XOR(db[59*N]), K02XOR(db[60*N]), &db[31*N], &db[11*N], &db[21*N], &db[ 6*N]);
			s6(K49XOR(db[(E0)+(32*N)]), K36XOR(db[(E1)+(32*N)]), K16XOR(db[(E2)+(32*N)]), K01XOR(db[(E3)+(32*N)]), K28XOR(db[(E4)+(32*N)]), K37XOR(db[(E5)+(32*N)]), &db[ 3*N], &db[28*N], &db[10*N], &db[18*N]); 
			__syncthreads();
			s7(K01XOR(db[23*N]), K23XOR(db[24*N]), K36XOR(db[25*N]), K03XOR(db[26*N]), K14XOR(db[27*N]), K24XOR(db[28*N]), &db[63*N], &db[43*N], &db[53*N], &db[38*N]);
			s6(K42XOR(db[E0]), K29XOR(db[E1]), K09XOR(db[E2]), K51XOR(db[E3]), K21XOR(db[E4]), K30XOR(db[E5]), &db[35*N], &db[60*N], &db[42*N], &db[50*N]); 
			break;
		}
		__syncthreads();
	}

#endif
}

#define GET_TRIPCODE_CHAR_INDEX(r, t, i0, i1, i2, i3, i4, i5, pos)  \
		(  ((((r)[threadIdx.x + (i0*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << (5 + ((pos) * 6)))  \
	 	 | ((((r)[threadIdx.x + (i1*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << (4 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i2*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << (3 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i3*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << (2 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i4*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << (1 + ((pos) * 6)))  \
		 | ((((r)[threadIdx.x + (i5*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << (0 + ((pos) * 6)))) \

#define GET_TRIPCODE_CHAR_INDEX_LAST(r, t, i0, i1, i2, i3)     \
		(  ((((r)[threadIdx.x + (i0*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << 5)  \
	 	 | ((((r)[threadIdx.x + (i1*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << 4)  \
		 | ((((r)[threadIdx.x + (i2*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << 3)  \
		 | ((((r)[threadIdx.x + (i3*N)] & (0x01 << (t))) ? (0x1) : (0x0)) << 2)) \

DES_FUNCTION_QUALIFIERS void
DES_GetTripcodeChunks(int tripcodeIndex, unsigned int *tripcodeChunkArray, int searchMode)
{
	// Perform the final permutation here.
	if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
	} else if (searchMode == SEARCH_MODE_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
		                        | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
		                        | GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
								| GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] = ((tripcodeChunkArray[0] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0);
		tripcodeChunkArray[2] = ((tripcodeChunkArray[1] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0);
		tripcodeChunkArray[3] = ((tripcodeChunkArray[2] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0);
		tripcodeChunkArray[4] = ((tripcodeChunkArray[3] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0);
		tripcodeChunkArray[5] = ((tripcodeChunkArray[4] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	}
}

DES_FUNCTION_QUALIFIERS
unsigned char *DES_GetTripcode(int tripcodeIndex, unsigned char *tripcode)
{
	// Perform the final permutation as necessary.
  	tripcode[0] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0)];
  	tripcode[1] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0)];
  	tripcode[2] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0)];
  	tripcode[3] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0)];
  	tripcode[4] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0)];
  	tripcode[5] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0)];
  	tripcode[6] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0)];
  	tripcode[7] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0)];
  	tripcode[8] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0)];
	tripcode[9] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24)];
 	tripcode[10] = '\0';

	return tripcode;
}

#define SET_KEY_CHAR(var, flag, table, value)             \
	if (!(flag)) {                                        \
		var = (table)[(value)];                           \
		isSecondByte = IS_FIRST_BYTE_SJIS(var);           \
	} else {                                              \
		var = cudaKeyCharTable_SecondByte[(value)];          \
		isSecondByte = FALSE;                             \
	}

#define CUDA_DES_DEFINE_SEARCH_FUNCTION(functionName) \
__global__ void functionName(\
	GPUOutput *outputArray,\
	unsigned char      *chunkBitmap,\
	unsigned int     *tripcodeChunkArray,\
	unsigned int      numTripcodeChunk,\
	unsigned char   *CUDA_key,\
	unsigned char   *CUDA_expansionFunction,\
	unsigned char   *CUDA_key0Array,\
	unsigned char   *CUDA_key7Array,\
	DES_Vector      *CUDA_keyFrom49To55Array,\
	int         searchMode) {

#define CUDA_DES_BEFORE_SEARCHING \
	GPUOutput  *output = &outputArray[blockIdx.x * blockDim.x + threadIdx.x];\
	unsigned char        key[8];\
	BOOL         isSecondByte;\
	unsigned char        tripcodeIndex;\
	unsigned char        passCount = 0;\
	BOOL found = FALSE;\
	\
	if (threadIdx.y == 0) {\
		output->numMatchingTripcodes = 0;\
	}\
	key[1] = CUDA_key[1];\
	key[2] = CUDA_key[2];\
	\
	for (passCount = 0; passCount < CUDA_DES_MAX_PASS_COUNT; ++passCount) {\
		key[0] = CUDA_key0Array[passCount];\
		isSecondByte = IS_FIRST_BYTE_SJIS(CUDA_key[2]);\
		SET_KEY_CHAR(key[3], isSecondByte, cudaKeyCharTable_FirstByte, CUDA_key[3] + (((threadIdx.x >> 6) &  3) | ((blockIdx.x & (3 << 12)) >> (12 - 2))));\
		SET_KEY_CHAR(key[4], isSecondByte, cudaKeyCharTable_FirstByte, CUDA_key[4] + ( (blockIdx.x  >> 6) & 63));\
		SET_KEY_CHAR(key[5], isSecondByte, cudaKeyCharTable_FirstByte, CUDA_key[5] + (  blockIdx.x        & 63));\
		SET_KEY_CHAR(key[6], isSecondByte, cudaKeyCharTable_FirstByte, CUDA_key[6] + (  threadIdx.x       & 63));\
		unsigned int keyFrom00To27 = (((unsigned int)key[3] & 0x7f) << 21) | (((unsigned int)key[2] & 0x7f) << 14) | (((unsigned int)key[1] & 0x7f) <<  7) | (((unsigned int)key[0] & 0x7f) << 0); \
		unsigned int keyFrom28To48 = (((unsigned int)key[6] & 0x7f) << 14) | (((unsigned int)key[5] & 0x7f) <<  7) | (((unsigned int)key[4] & 0x7f) << 0); \
		__syncthreads();\
		DES_Crypt(keyFrom00To27, keyFrom28To48, CUDA_expansionFunction, CUDA_keyFrom49To55Array);\
		\
		__syncthreads();\
		if (threadIdx.y == 0) {\
			for (tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {

#define CUDA_DES_END_OF_SEAERCH_FUNCTION \
			}\
		}\
	}\
quit_loops:\
	if (found == TRUE) {\
		output->numMatchingTripcodes  = 1;\
		output->pair.key.c[0] = key[0];\
		output->pair.key.c[1] = key[1];\
		output->pair.key.c[2] = key[2];\
		output->pair.key.c[3] = key[3];\
		output->pair.key.c[4] = key[4];\
		output->pair.key.c[5] = key[5];\
		output->pair.key.c[6] = key[6];\
		output->pair.key.c[7] = CUDA_key7Array[tripcodeIndex];\
	}\
	if (threadIdx.y == 0)\
		output->numGeneratedTripcodes = CUDA_DES_BS_DEPTH * passCount;\
}

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	if (CUDA_smallChunkBitmap[tripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)])
		continue;
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == tripcodeChunk) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	if (CUDA_smallChunkBitmap[tripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)] || chunkBitmap[tripcodeChunk >> ((5 - CHUNK_BITMAP_LEN_STRING) * 6)])
		continue;
	int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
	while (tripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
		middle = (lower + upper) >> 1;
		if (tripcodeChunk > tripcodeChunkArray[middle]) {
			lower = middle + 1;
		} else {
			upper = middle - 1;
		}
	}
	if (tripcodeChunk == tripcodeChunkArray[middle]) {
		found = TRUE;
		goto quit_loops;
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_BackwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX_LAST(dataBlocks, tripcodeIndex, 48, 16, 56, 24) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		if (CUDA_smallChunkBitmap[generatedTripcodeChunkArray[pos] >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)])
			continue;
		for (int j = 0; j < numTripcodeChunk; ++j){
			if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[pos]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		unsigned int generatedTripcodeChunk = generatedTripcodeChunkArray[pos];
		if (   CUDA_smallChunkBitmap[generatedTripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)] 
		    || chunkBitmap[generatedTripcodeChunk >> ((5 - CHUNK_BITMAP_LEN_STRING) * 6)])
			continue;
		int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
		while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
			middle = (lower + upper) >> 1;
			if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
				lower = middle + 1;
			} else {
				upper = middle - 1;
			}
		}
		if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	if (!CUDA_smallChunkBitmap[generatedTripcodeChunkArray[0] >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[0]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
	//
	if (!CUDA_smallChunkBitmap[generatedTripcodeChunkArray[1] >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[1]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching)
	unsigned int generatedTripcodeChunkArray[6];
	unsigned int generatedTripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[0];
	if (!CUDA_smallChunkBitmap[generatedTripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)] && !chunkBitmap[generatedTripcodeChunk >> ((5 - CHUNK_BITMAP_LEN_STRING) * 6)]) {
		int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
		while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
			middle = (lower + upper) >> 1;
			if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
				lower = middle + 1;
			} else {
				upper = middle - 1;
			}
		}
		if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
			found = TRUE;
			goto quit_loops;
		}
	}
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[1];
	if (!CUDA_smallChunkBitmap[generatedTripcodeChunk >> ((5 - SMALL_CHUNK_BITMAP_LEN_STRING) * 6)] && !chunkBitmap[generatedTripcodeChunk >> ((5 - CHUNK_BITMAP_LEN_STRING) * 6)]) {
		int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
		while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
			middle = (lower + upper) >> 1;
			if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
				lower = middle + 1;
			} else {
				upper = middle - 1;
			}
		}
		if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION



///////////////////////////////////////////////////////////////////////////////
// CUDA SEARCH THREAD FOR 10 CHARACTER TRIPCODES                             //
///////////////////////////////////////////////////////////////////////////////

#define SET_BIT_FOR_KEY7(var, k) if (key7 & (0x1 << (k))) (var) |= 0x1 << tripcodeIndex

unsigned WINAPI Thread_SearchForDESTripcodesOnCUDADevice(LPVOID info)
{
	hipDeviceProp_t  CUDADeviceProperties;
	unsigned int    numBlocksPerSM;
	unsigned int    numBlocksPerGrid;
	GPUOutput      *outputArray = NULL;
	GPUOutput      *CUDA_outputArray = NULL;
	unsigned int   *CUDA_tripcodeChunkArray = NULL;
	unsigned char  *CUDA_chunkBitmap = NULL;
	unsigned int    sizeOutputArray;
	unsigned char   key[MAX_LEN_TRIPCODE + 1];
	unsigned char   expansionFunction[96];
	char            status[LEN_LINE_BUFFER_FOR_SCREEN] = "";
	double          timeElapsed = 0;
	double          numGeneratedTripcodes = 0;
	double          speed = 0;
	DWORD           startingTime;
	DWORD           endingTime;
	double          deltaTime;

	unsigned char   *CUDA_key; // [12];
	unsigned char   *CUDA_expansionFunction; // [96];
	unsigned char   *CUDA_key0Array; // [CUDA_DES_MAX_PASS_COUNT];
	unsigned char   *CUDA_key7Array; // [CUDA_DES_BS_DEPTH];
	DES_Vector      *CUDA_keyFrom49To55Array; // [7];

	key[lenTripcode] = '\0';
	
	CUDA_ERROR(hipSetDevice(((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	CUDA_ERROR(hipGetDeviceProperties(&CUDADeviceProperties, ((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	if (CUDADeviceProperties.computeMode == hipComputeModeProhibited) {
		sprintf(status, "[disabled]");
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), status);
		return 0;
	}
	int numThreadsPerBlock = (CUDADeviceProperties.major == 3 && CUDADeviceProperties.minor == 7) ? 448 :
		                     (CUDADeviceProperties.major == 2                                   ) ? 768 :
		                     (CUDADeviceProperties.major == 3                                   ) ? 768 :
		                     (CUDADeviceProperties.major == 5                                   ) ? 512 :
		                                                                                            512;
	int numBitsliceDESPerBlock = numThreadsPerBlock / NUM_THREADS_PER_BITSICE_DES;

	numBlocksPerSM = options.CUDANumBlocksPerSM;
	numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
	sizeOutputArray = numBitsliceDESPerBlock * numBlocksPerGrid;
	outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(outputArray == NULL, ERROR_NO_MEMORY, GetErrorMessage(ERROR_NO_MEMORY));
	CUDA_ERROR(hipMalloc((void **)&CUDA_outputArray,        sizeof(GPUOutput) * sizeOutputArray));
	CUDA_ERROR(hipMalloc((void **)&CUDA_chunkBitmap,        CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMalloc((void **)&CUDA_tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk)); 
	CUDA_ERROR(hipMalloc((void **)&CUDA_key,                sizeof(unsigned char) * 8)); 
	CUDA_ERROR(hipMalloc((void **)&CUDA_expansionFunction,  sizeof(unsigned char) * 96)); 
	CUDA_ERROR(hipMalloc((void **)&CUDA_key0Array,           sizeof(unsigned char) * CUDA_DES_MAX_PASS_COUNT)); 
	CUDA_ERROR(hipMalloc((void **)&CUDA_key7Array,          sizeof(unsigned char) * CUDA_DES_BS_DEPTH)); 
	CUDA_ERROR(hipMalloc((void **)&CUDA_keyFrom49To55Array, sizeof(DES_Vector)    * 7)); 

	EnterCriticalSection(&((CUDADeviceSearchThreadInfo *)info)->criticalSection);
	CUDA_ERROR(hipMemcpy(CUDA_tripcodeChunkArray, tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpy(CUDA_chunkBitmap, chunkBitmap, CHUNK_BITMAP_SIZE, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_smallChunkBitmap), smallChunkBitmap, SMALL_CHUNK_BITMAP_SIZE));
	LeaveCriticalSection(&((CUDADeviceSearchThreadInfo *)info)->criticalSection);
	
	startingTime = timeGetTime();

	while (!GetTerminationState()) {
		// Choose the first 3 characters of the key.
		SetCharactersInTripcodeKey(key, 3);
		unsigned char  salt[2];
		salt[0] = CONVERT_CHAR_FOR_SALT(key[1]);
		salt[1] = CONVERT_CHAR_FOR_SALT(key[2]);
		
		//
		unsigned char key0Array[CUDA_DES_MAX_PASS_COUNT];
		unsigned char randomByteForKey0 = RandomByte();
		int j = 0;
		for (int i = 3; i < lenTripcode; ++i)
			key[i] = 'A';
		for (int i = 0; i < CUDA_DES_MAX_PASS_COUNT; ++i) {
			do {
				key[0] = keyCharTable_FirstByte[randomByteForKey0 + j++];
			} while(!IsValidKey(key));
			key0Array[i] = key[0];
		}

		// Generate random bytes for the key to ensure its randomness.
		for (int i = 3; i < lenTripcode; ++i)
			key[i] = RandomByte();
		
		//
		unsigned char key7Array[CUDA_DES_BS_DEPTH];
		DES_Vector  keyFrom49To55Array[7] = {0, 0, 0, 0, 0, 0, 0};
		for (int tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {
			unsigned char key7 = key7Array[tripcodeIndex] = keyCharTable_SecondByteAndOneByte[key[7] + tripcodeIndex];
			SET_BIT_FOR_KEY7(keyFrom49To55Array[0], 0);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[1], 1);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[2], 2);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[3], 3);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[4], 4);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[5], 5);
			SET_BIT_FOR_KEY7(keyFrom49To55Array[6], 6);
		}

		// Create an expansion function based on the salt.
		salt[0] = CONVERT_CHAR_FOR_SALT(key[1]);
		salt[1] = CONVERT_CHAR_FOR_SALT(key[2]);
		DES_CreateExpansionFunction((char *)salt, expansionFunction);

		// Call an appropriate CUDA kernel.
		CUDA_ERROR(hipMemcpy(CUDA_key,               key,               8, hipMemcpyHostToDevice));
		CUDA_ERROR(hipMemcpy(CUDA_expansionFunction, expansionFunction, sizeof(expansionFunction), hipMemcpyHostToDevice));
		CUDA_ERROR(hipMemcpy(CUDA_key0Array,         key0Array,         sizeof(key0Array), hipMemcpyHostToDevice));
		CUDA_ERROR(hipMemcpy(CUDA_key7Array,         key7Array,         sizeof(key7Array), hipMemcpyHostToDevice));
		CUDA_ERROR(hipMemcpy(CUDA_keyFrom49To55Array, keyFrom49To55Array, sizeof(keyFrom49To55Array), hipMemcpyHostToDevice));
		dim3 dimBlock(numBitsliceDESPerBlock, NUM_THREADS_PER_BITSICE_DES);
		dim3 dimGrid(numBlocksPerGrid);
		if (searchMode == SEARCH_MODE_FLEXIBLE) {
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_Flexible_Simple<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					CUDA_key,
					CUDA_expansionFunction,
					CUDA_key0Array,
					CUDA_key7Array,
					CUDA_keyFrom49To55Array,
					searchMode);
			} else {
				CUDA_PerformSearching_DES_Flexible<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					CUDA_key,
					CUDA_expansionFunction,
					CUDA_key0Array,
					CUDA_key7Array,
					CUDA_keyFrom49To55Array,
					searchMode);
			}
		} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					CUDA_key,
					CUDA_expansionFunction,
					CUDA_key0Array,
					CUDA_key7Array,
					CUDA_keyFrom49To55Array,
					searchMode);
			} else {
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					CUDA_key,
					CUDA_expansionFunction,
					CUDA_key0Array,
					CUDA_key7Array,
					CUDA_keyFrom49To55Array,
					searchMode);
			}
		} else {
			if (numTripcodeChunk == 1) {
				if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
					CUDA_PerformSearching_DES_ForwardMatching_1Chunk<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
						CUDA_outputArray,
						CUDA_chunkBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						CUDA_key,
						CUDA_expansionFunction,
						CUDA_key0Array,
						CUDA_key7Array,
						CUDA_keyFrom49To55Array,
						searchMode);
				} else {
					CUDA_PerformSearching_DES_BackwardMatching_1Chunk<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
						CUDA_outputArray,
						CUDA_chunkBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						CUDA_key,
						CUDA_expansionFunction,
						CUDA_key0Array,
						CUDA_key7Array,
						CUDA_keyFrom49To55Array,
						searchMode);
				}
			} else if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					CUDA_key,
					CUDA_expansionFunction,
					CUDA_key0Array,
					CUDA_key7Array,
					CUDA_keyFrom49To55Array,
					searchMode);
			} else {
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching<<<dimGrid, dimBlock, CUDADeviceProperties.sharedMemPerBlock>>>(
					CUDA_outputArray,
					CUDA_chunkBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					CUDA_key,
					CUDA_expansionFunction,
					CUDA_key0Array,
					CUDA_key7Array,
					CUDA_keyFrom49To55Array,
					searchMode);
			}
		}
		CUDA_ERROR(hipGetLastError());
		// CUDA_ERROR(hipDeviceSynchronize()); // Check errors at kernel launch.

		// Process the output array.
		CUDA_ERROR(hipMemcpy(outputArray, CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray, hipMemcpyDeviceToHost));
		// We can save registers this way.
		for (unsigned int indexOutput = 0; indexOutput < sizeOutputArray; indexOutput++){
			GPUOutput *output = &outputArray[indexOutput];
			if (output->numMatchingTripcodes > 0)
				GenerateDESTripcode(output->pair.tripcode.c, output->pair.key.c);
		}
		numGeneratedTripcodes += ProcessGPUOutput(key, outputArray, sizeOutputArray, FALSE);
		
		//
		endingTime = timeGetTime();
		deltaTime = (endingTime >= startingTime)
						? ((double)endingTime - (double)startingTime                     ) * 0.001
						: ((double)endingTime - (double)startingTime + (double)0xffffffff) * 0.001;
		while (GetPauseState() && !GetTerminationState())
			Sleep(PAUSE_INTERVAL);
		startingTime = timeGetTime();
		timeElapsed += deltaTime;
		speed = numGeneratedTripcodes / timeElapsed;
		//
		sprintf(status,
			    "%.1lfM TPS, %d blocks/SM",
				speed / 1000000,
				numBlocksPerSM);
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), status);
	}

	RELEASE_AND_SET_TO_NULL(CUDA_outputArray,        hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_tripcodeChunkArray, hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_chunkBitmap,        hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_key,                hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_expansionFunction,  hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_key0Array,          hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_key7Array,          hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_keyFrom49To55Array, hipFree);
	RELEASE_AND_SET_TO_NULL(outputArray,             free);

	return 0;
}
