// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 2048
#include "CUDA10_Registers_Kernel.h"
#define SALT 2049
#include "CUDA10_Registers_Kernel.h"
#define SALT 2050
#include "CUDA10_Registers_Kernel.h"
#define SALT 2051
#include "CUDA10_Registers_Kernel.h"
#define SALT 2052
#include "CUDA10_Registers_Kernel.h"
#define SALT 2053
#include "CUDA10_Registers_Kernel.h"
#define SALT 2054
#include "CUDA10_Registers_Kernel.h"
#define SALT 2055
#include "CUDA10_Registers_Kernel.h"
#define SALT 2056
#include "CUDA10_Registers_Kernel.h"
#define SALT 2057
#include "CUDA10_Registers_Kernel.h"
#define SALT 2058
#include "CUDA10_Registers_Kernel.h"
#define SALT 2059
#include "CUDA10_Registers_Kernel.h"
#define SALT 2060
#include "CUDA10_Registers_Kernel.h"
#define SALT 2061
#include "CUDA10_Registers_Kernel.h"
#define SALT 2062
#include "CUDA10_Registers_Kernel.h"
#define SALT 2063
#include "CUDA10_Registers_Kernel.h"
#define SALT 2064
#include "CUDA10_Registers_Kernel.h"
#define SALT 2065
#include "CUDA10_Registers_Kernel.h"
#define SALT 2066
#include "CUDA10_Registers_Kernel.h"
#define SALT 2067
#include "CUDA10_Registers_Kernel.h"
#define SALT 2068
#include "CUDA10_Registers_Kernel.h"
#define SALT 2069
#include "CUDA10_Registers_Kernel.h"
#define SALT 2070
#include "CUDA10_Registers_Kernel.h"
#define SALT 2071
#include "CUDA10_Registers_Kernel.h"
#define SALT 2072
#include "CUDA10_Registers_Kernel.h"
#define SALT 2073
#include "CUDA10_Registers_Kernel.h"
#define SALT 2074
#include "CUDA10_Registers_Kernel.h"
#define SALT 2075
#include "CUDA10_Registers_Kernel.h"
#define SALT 2076
#include "CUDA10_Registers_Kernel.h"
#define SALT 2077
#include "CUDA10_Registers_Kernel.h"
#define SALT 2078
#include "CUDA10_Registers_Kernel.h"
#define SALT 2079
#include "CUDA10_Registers_Kernel.h"
#define SALT 2080
#include "CUDA10_Registers_Kernel.h"
#define SALT 2081
#include "CUDA10_Registers_Kernel.h"
#define SALT 2082
#include "CUDA10_Registers_Kernel.h"
#define SALT 2083
#include "CUDA10_Registers_Kernel.h"
#define SALT 2084
#include "CUDA10_Registers_Kernel.h"
#define SALT 2085
#include "CUDA10_Registers_Kernel.h"
#define SALT 2086
#include "CUDA10_Registers_Kernel.h"
#define SALT 2087
#include "CUDA10_Registers_Kernel.h"
#define SALT 2088
#include "CUDA10_Registers_Kernel.h"
#define SALT 2089
#include "CUDA10_Registers_Kernel.h"
#define SALT 2090
#include "CUDA10_Registers_Kernel.h"
#define SALT 2091
#include "CUDA10_Registers_Kernel.h"
#define SALT 2092
#include "CUDA10_Registers_Kernel.h"
#define SALT 2093
#include "CUDA10_Registers_Kernel.h"
#define SALT 2094
#include "CUDA10_Registers_Kernel.h"
#define SALT 2095
#include "CUDA10_Registers_Kernel.h"
#define SALT 2096
#include "CUDA10_Registers_Kernel.h"
#define SALT 2097
#include "CUDA10_Registers_Kernel.h"
#define SALT 2098
#include "CUDA10_Registers_Kernel.h"
#define SALT 2099
#include "CUDA10_Registers_Kernel.h"
#define SALT 2100
#include "CUDA10_Registers_Kernel.h"
#define SALT 2101
#include "CUDA10_Registers_Kernel.h"
#define SALT 2102
#include "CUDA10_Registers_Kernel.h"
#define SALT 2103
#include "CUDA10_Registers_Kernel.h"
#define SALT 2104
#include "CUDA10_Registers_Kernel.h"
#define SALT 2105
#include "CUDA10_Registers_Kernel.h"
#define SALT 2106
#include "CUDA10_Registers_Kernel.h"
#define SALT 2107
#include "CUDA10_Registers_Kernel.h"
#define SALT 2108
#include "CUDA10_Registers_Kernel.h"
#define SALT 2109
#include "CUDA10_Registers_Kernel.h"
#define SALT 2110
#include "CUDA10_Registers_Kernel.h"
#define SALT 2111
#include "CUDA10_Registers_Kernel.h"
#define SALT 2112
#include "CUDA10_Registers_Kernel.h"
#define SALT 2113
#include "CUDA10_Registers_Kernel.h"
#define SALT 2114
#include "CUDA10_Registers_Kernel.h"
#define SALT 2115
#include "CUDA10_Registers_Kernel.h"
#define SALT 2116
#include "CUDA10_Registers_Kernel.h"
#define SALT 2117
#include "CUDA10_Registers_Kernel.h"
#define SALT 2118
#include "CUDA10_Registers_Kernel.h"
#define SALT 2119
#include "CUDA10_Registers_Kernel.h"
#define SALT 2120
#include "CUDA10_Registers_Kernel.h"
#define SALT 2121
#include "CUDA10_Registers_Kernel.h"
#define SALT 2122
#include "CUDA10_Registers_Kernel.h"
#define SALT 2123
#include "CUDA10_Registers_Kernel.h"
#define SALT 2124
#include "CUDA10_Registers_Kernel.h"
#define SALT 2125
#include "CUDA10_Registers_Kernel.h"
#define SALT 2126
#include "CUDA10_Registers_Kernel.h"
#define SALT 2127
#include "CUDA10_Registers_Kernel.h"
#define SALT 2128
#include "CUDA10_Registers_Kernel.h"
#define SALT 2129
#include "CUDA10_Registers_Kernel.h"
#define SALT 2130
#include "CUDA10_Registers_Kernel.h"
#define SALT 2131
#include "CUDA10_Registers_Kernel.h"
#define SALT 2132
#include "CUDA10_Registers_Kernel.h"
#define SALT 2133
#include "CUDA10_Registers_Kernel.h"
#define SALT 2134
#include "CUDA10_Registers_Kernel.h"
#define SALT 2135
#include "CUDA10_Registers_Kernel.h"
#define SALT 2136
#include "CUDA10_Registers_Kernel.h"
#define SALT 2137
#include "CUDA10_Registers_Kernel.h"
#define SALT 2138
#include "CUDA10_Registers_Kernel.h"
#define SALT 2139
#include "CUDA10_Registers_Kernel.h"
#define SALT 2140
#include "CUDA10_Registers_Kernel.h"
#define SALT 2141
#include "CUDA10_Registers_Kernel.h"
#define SALT 2142
#include "CUDA10_Registers_Kernel.h"
#define SALT 2143
#include "CUDA10_Registers_Kernel.h"
#define SALT 2144
#include "CUDA10_Registers_Kernel.h"
#define SALT 2145
#include "CUDA10_Registers_Kernel.h"
#define SALT 2146
#include "CUDA10_Registers_Kernel.h"
#define SALT 2147
#include "CUDA10_Registers_Kernel.h"
#define SALT 2148
#include "CUDA10_Registers_Kernel.h"
#define SALT 2149
#include "CUDA10_Registers_Kernel.h"
#define SALT 2150
#include "CUDA10_Registers_Kernel.h"
#define SALT 2151
#include "CUDA10_Registers_Kernel.h"
#define SALT 2152
#include "CUDA10_Registers_Kernel.h"
#define SALT 2153
#include "CUDA10_Registers_Kernel.h"
#define SALT 2154
#include "CUDA10_Registers_Kernel.h"
#define SALT 2155
#include "CUDA10_Registers_Kernel.h"
#define SALT 2156
#include "CUDA10_Registers_Kernel.h"
#define SALT 2157
#include "CUDA10_Registers_Kernel.h"
#define SALT 2158
#include "CUDA10_Registers_Kernel.h"
#define SALT 2159
#include "CUDA10_Registers_Kernel.h"
#define SALT 2160
#include "CUDA10_Registers_Kernel.h"
#define SALT 2161
#include "CUDA10_Registers_Kernel.h"
#define SALT 2162
#include "CUDA10_Registers_Kernel.h"
#define SALT 2163
#include "CUDA10_Registers_Kernel.h"
#define SALT 2164
#include "CUDA10_Registers_Kernel.h"
#define SALT 2165
#include "CUDA10_Registers_Kernel.h"
#define SALT 2166
#include "CUDA10_Registers_Kernel.h"
#define SALT 2167
#include "CUDA10_Registers_Kernel.h"
#define SALT 2168
#include "CUDA10_Registers_Kernel.h"
#define SALT 2169
#include "CUDA10_Registers_Kernel.h"
#define SALT 2170
#include "CUDA10_Registers_Kernel.h"
#define SALT 2171
#include "CUDA10_Registers_Kernel.h"
#define SALT 2172
#include "CUDA10_Registers_Kernel.h"
#define SALT 2173
#include "CUDA10_Registers_Kernel.h"
#define SALT 2174
#include "CUDA10_Registers_Kernel.h"
#define SALT 2175
#include "CUDA10_Registers_Kernel.h"
#define SALT 2176
#include "CUDA10_Registers_Kernel.h"
#define SALT 2177
#include "CUDA10_Registers_Kernel.h"
#define SALT 2178
#include "CUDA10_Registers_Kernel.h"
#define SALT 2179
#include "CUDA10_Registers_Kernel.h"
#define SALT 2180
#include "CUDA10_Registers_Kernel.h"
#define SALT 2181
#include "CUDA10_Registers_Kernel.h"
#define SALT 2182
#include "CUDA10_Registers_Kernel.h"
#define SALT 2183
#include "CUDA10_Registers_Kernel.h"
#define SALT 2184
#include "CUDA10_Registers_Kernel.h"
#define SALT 2185
#include "CUDA10_Registers_Kernel.h"
#define SALT 2186
#include "CUDA10_Registers_Kernel.h"
#define SALT 2187
#include "CUDA10_Registers_Kernel.h"
#define SALT 2188
#include "CUDA10_Registers_Kernel.h"
#define SALT 2189
#include "CUDA10_Registers_Kernel.h"
#define SALT 2190
#include "CUDA10_Registers_Kernel.h"
#define SALT 2191
#include "CUDA10_Registers_Kernel.h"
#define SALT 2192
#include "CUDA10_Registers_Kernel.h"
#define SALT 2193
#include "CUDA10_Registers_Kernel.h"
#define SALT 2194
#include "CUDA10_Registers_Kernel.h"
#define SALT 2195
#include "CUDA10_Registers_Kernel.h"
#define SALT 2196
#include "CUDA10_Registers_Kernel.h"
#define SALT 2197
#include "CUDA10_Registers_Kernel.h"
#define SALT 2198
#include "CUDA10_Registers_Kernel.h"
#define SALT 2199
#include "CUDA10_Registers_Kernel.h"
#define SALT 2200
#include "CUDA10_Registers_Kernel.h"
#define SALT 2201
#include "CUDA10_Registers_Kernel.h"
#define SALT 2202
#include "CUDA10_Registers_Kernel.h"
#define SALT 2203
#include "CUDA10_Registers_Kernel.h"
#define SALT 2204
#include "CUDA10_Registers_Kernel.h"
#define SALT 2205
#include "CUDA10_Registers_Kernel.h"
#define SALT 2206
#include "CUDA10_Registers_Kernel.h"
#define SALT 2207
#include "CUDA10_Registers_Kernel.h"
#define SALT 2208
#include "CUDA10_Registers_Kernel.h"
#define SALT 2209
#include "CUDA10_Registers_Kernel.h"
#define SALT 2210
#include "CUDA10_Registers_Kernel.h"
#define SALT 2211
#include "CUDA10_Registers_Kernel.h"
#define SALT 2212
#include "CUDA10_Registers_Kernel.h"
#define SALT 2213
#include "CUDA10_Registers_Kernel.h"
#define SALT 2214
#include "CUDA10_Registers_Kernel.h"
#define SALT 2215
#include "CUDA10_Registers_Kernel.h"
#define SALT 2216
#include "CUDA10_Registers_Kernel.h"
#define SALT 2217
#include "CUDA10_Registers_Kernel.h"
#define SALT 2218
#include "CUDA10_Registers_Kernel.h"
#define SALT 2219
#include "CUDA10_Registers_Kernel.h"
#define SALT 2220
#include "CUDA10_Registers_Kernel.h"
#define SALT 2221
#include "CUDA10_Registers_Kernel.h"
#define SALT 2222
#include "CUDA10_Registers_Kernel.h"
#define SALT 2223
#include "CUDA10_Registers_Kernel.h"
#define SALT 2224
#include "CUDA10_Registers_Kernel.h"
#define SALT 2225
#include "CUDA10_Registers_Kernel.h"
#define SALT 2226
#include "CUDA10_Registers_Kernel.h"
#define SALT 2227
#include "CUDA10_Registers_Kernel.h"
#define SALT 2228
#include "CUDA10_Registers_Kernel.h"
#define SALT 2229
#include "CUDA10_Registers_Kernel.h"
#define SALT 2230
#include "CUDA10_Registers_Kernel.h"
#define SALT 2231
#include "CUDA10_Registers_Kernel.h"
#define SALT 2232
#include "CUDA10_Registers_Kernel.h"
#define SALT 2233
#include "CUDA10_Registers_Kernel.h"
#define SALT 2234
#include "CUDA10_Registers_Kernel.h"
#define SALT 2235
#include "CUDA10_Registers_Kernel.h"
#define SALT 2236
#include "CUDA10_Registers_Kernel.h"
#define SALT 2237
#include "CUDA10_Registers_Kernel.h"
#define SALT 2238
#include "CUDA10_Registers_Kernel.h"
#define SALT 2239
#include "CUDA10_Registers_Kernel.h"
#define SALT 2240
#include "CUDA10_Registers_Kernel.h"
#define SALT 2241
#include "CUDA10_Registers_Kernel.h"
#define SALT 2242
#include "CUDA10_Registers_Kernel.h"
#define SALT 2243
#include "CUDA10_Registers_Kernel.h"
#define SALT 2244
#include "CUDA10_Registers_Kernel.h"
#define SALT 2245
#include "CUDA10_Registers_Kernel.h"
#define SALT 2246
#include "CUDA10_Registers_Kernel.h"
#define SALT 2247
#include "CUDA10_Registers_Kernel.h"
#define SALT 2248
#include "CUDA10_Registers_Kernel.h"
#define SALT 2249
#include "CUDA10_Registers_Kernel.h"
#define SALT 2250
#include "CUDA10_Registers_Kernel.h"
#define SALT 2251
#include "CUDA10_Registers_Kernel.h"
#define SALT 2252
#include "CUDA10_Registers_Kernel.h"
#define SALT 2253
#include "CUDA10_Registers_Kernel.h"
#define SALT 2254
#include "CUDA10_Registers_Kernel.h"
#define SALT 2255
#include "CUDA10_Registers_Kernel.h"
#define SALT 2256
#include "CUDA10_Registers_Kernel.h"
#define SALT 2257
#include "CUDA10_Registers_Kernel.h"
#define SALT 2258
#include "CUDA10_Registers_Kernel.h"
#define SALT 2259
#include "CUDA10_Registers_Kernel.h"
#define SALT 2260
#include "CUDA10_Registers_Kernel.h"
#define SALT 2261
#include "CUDA10_Registers_Kernel.h"
#define SALT 2262
#include "CUDA10_Registers_Kernel.h"
#define SALT 2263
#include "CUDA10_Registers_Kernel.h"
#define SALT 2264
#include "CUDA10_Registers_Kernel.h"
#define SALT 2265
#include "CUDA10_Registers_Kernel.h"
#define SALT 2266
#include "CUDA10_Registers_Kernel.h"
#define SALT 2267
#include "CUDA10_Registers_Kernel.h"
#define SALT 2268
#include "CUDA10_Registers_Kernel.h"
#define SALT 2269
#include "CUDA10_Registers_Kernel.h"
#define SALT 2270
#include "CUDA10_Registers_Kernel.h"
#define SALT 2271
#include "CUDA10_Registers_Kernel.h"
#define SALT 2272
#include "CUDA10_Registers_Kernel.h"
#define SALT 2273
#include "CUDA10_Registers_Kernel.h"
#define SALT 2274
#include "CUDA10_Registers_Kernel.h"
#define SALT 2275
#include "CUDA10_Registers_Kernel.h"
#define SALT 2276
#include "CUDA10_Registers_Kernel.h"
#define SALT 2277
#include "CUDA10_Registers_Kernel.h"
#define SALT 2278
#include "CUDA10_Registers_Kernel.h"
#define SALT 2279
#include "CUDA10_Registers_Kernel.h"
#define SALT 2280
#include "CUDA10_Registers_Kernel.h"
#define SALT 2281
#include "CUDA10_Registers_Kernel.h"
#define SALT 2282
#include "CUDA10_Registers_Kernel.h"
#define SALT 2283
#include "CUDA10_Registers_Kernel.h"
#define SALT 2284
#include "CUDA10_Registers_Kernel.h"
#define SALT 2285
#include "CUDA10_Registers_Kernel.h"
#define SALT 2286
#include "CUDA10_Registers_Kernel.h"
#define SALT 2287
#include "CUDA10_Registers_Kernel.h"
#define SALT 2288
#include "CUDA10_Registers_Kernel.h"
#define SALT 2289
#include "CUDA10_Registers_Kernel.h"
#define SALT 2290
#include "CUDA10_Registers_Kernel.h"
#define SALT 2291
#include "CUDA10_Registers_Kernel.h"
#define SALT 2292
#include "CUDA10_Registers_Kernel.h"
#define SALT 2293
#include "CUDA10_Registers_Kernel.h"
#define SALT 2294
#include "CUDA10_Registers_Kernel.h"
#define SALT 2295
#include "CUDA10_Registers_Kernel.h"
#define SALT 2296
#include "CUDA10_Registers_Kernel.h"
#define SALT 2297
#include "CUDA10_Registers_Kernel.h"
#define SALT 2298
#include "CUDA10_Registers_Kernel.h"
#define SALT 2299
#include "CUDA10_Registers_Kernel.h"
#define SALT 2300
#include "CUDA10_Registers_Kernel.h"
#define SALT 2301
#include "CUDA10_Registers_Kernel.h"
#define SALT 2302
#include "CUDA10_Registers_Kernel.h"
#define SALT 2303
#include "CUDA10_Registers_Kernel.h"
#define SALT 2304
#include "CUDA10_Registers_Kernel.h"
#define SALT 2305
#include "CUDA10_Registers_Kernel.h"
#define SALT 2306
#include "CUDA10_Registers_Kernel.h"
#define SALT 2307
#include "CUDA10_Registers_Kernel.h"
#define SALT 2308
#include "CUDA10_Registers_Kernel.h"
#define SALT 2309
#include "CUDA10_Registers_Kernel.h"
#define SALT 2310
#include "CUDA10_Registers_Kernel.h"
#define SALT 2311
#include "CUDA10_Registers_Kernel.h"
#define SALT 2312
#include "CUDA10_Registers_Kernel.h"
#define SALT 2313
#include "CUDA10_Registers_Kernel.h"
#define SALT 2314
#include "CUDA10_Registers_Kernel.h"
#define SALT 2315
#include "CUDA10_Registers_Kernel.h"
#define SALT 2316
#include "CUDA10_Registers_Kernel.h"
#define SALT 2317
#include "CUDA10_Registers_Kernel.h"
#define SALT 2318
#include "CUDA10_Registers_Kernel.h"
#define SALT 2319
#include "CUDA10_Registers_Kernel.h"
#define SALT 2320
#include "CUDA10_Registers_Kernel.h"
#define SALT 2321
#include "CUDA10_Registers_Kernel.h"
#define SALT 2322
#include "CUDA10_Registers_Kernel.h"
#define SALT 2323
#include "CUDA10_Registers_Kernel.h"
#define SALT 2324
#include "CUDA10_Registers_Kernel.h"
#define SALT 2325
#include "CUDA10_Registers_Kernel.h"
#define SALT 2326
#include "CUDA10_Registers_Kernel.h"
#define SALT 2327
#include "CUDA10_Registers_Kernel.h"
#define SALT 2328
#include "CUDA10_Registers_Kernel.h"
#define SALT 2329
#include "CUDA10_Registers_Kernel.h"
#define SALT 2330
#include "CUDA10_Registers_Kernel.h"
#define SALT 2331
#include "CUDA10_Registers_Kernel.h"
#define SALT 2332
#include "CUDA10_Registers_Kernel.h"
#define SALT 2333
#include "CUDA10_Registers_Kernel.h"
#define SALT 2334
#include "CUDA10_Registers_Kernel.h"
#define SALT 2335
#include "CUDA10_Registers_Kernel.h"
#define SALT 2336
#include "CUDA10_Registers_Kernel.h"
#define SALT 2337
#include "CUDA10_Registers_Kernel.h"
#define SALT 2338
#include "CUDA10_Registers_Kernel.h"
#define SALT 2339
#include "CUDA10_Registers_Kernel.h"
#define SALT 2340
#include "CUDA10_Registers_Kernel.h"
#define SALT 2341
#include "CUDA10_Registers_Kernel.h"
#define SALT 2342
#include "CUDA10_Registers_Kernel.h"
#define SALT 2343
#include "CUDA10_Registers_Kernel.h"
#define SALT 2344
#include "CUDA10_Registers_Kernel.h"
#define SALT 2345
#include "CUDA10_Registers_Kernel.h"
#define SALT 2346
#include "CUDA10_Registers_Kernel.h"
#define SALT 2347
#include "CUDA10_Registers_Kernel.h"
#define SALT 2348
#include "CUDA10_Registers_Kernel.h"
#define SALT 2349
#include "CUDA10_Registers_Kernel.h"
#define SALT 2350
#include "CUDA10_Registers_Kernel.h"
#define SALT 2351
#include "CUDA10_Registers_Kernel.h"
#define SALT 2352
#include "CUDA10_Registers_Kernel.h"
#define SALT 2353
#include "CUDA10_Registers_Kernel.h"
#define SALT 2354
#include "CUDA10_Registers_Kernel.h"
#define SALT 2355
#include "CUDA10_Registers_Kernel.h"
#define SALT 2356
#include "CUDA10_Registers_Kernel.h"
#define SALT 2357
#include "CUDA10_Registers_Kernel.h"
#define SALT 2358
#include "CUDA10_Registers_Kernel.h"
#define SALT 2359
#include "CUDA10_Registers_Kernel.h"
#define SALT 2360
#include "CUDA10_Registers_Kernel.h"
#define SALT 2361
#include "CUDA10_Registers_Kernel.h"
#define SALT 2362
#include "CUDA10_Registers_Kernel.h"
#define SALT 2363
#include "CUDA10_Registers_Kernel.h"
#define SALT 2364
#include "CUDA10_Registers_Kernel.h"
#define SALT 2365
#include "CUDA10_Registers_Kernel.h"
#define SALT 2366
#include "CUDA10_Registers_Kernel.h"
#define SALT 2367
#include "CUDA10_Registers_Kernel.h"
#define SALT 2368
#include "CUDA10_Registers_Kernel.h"
#define SALT 2369
#include "CUDA10_Registers_Kernel.h"
#define SALT 2370
#include "CUDA10_Registers_Kernel.h"
#define SALT 2371
#include "CUDA10_Registers_Kernel.h"
#define SALT 2372
#include "CUDA10_Registers_Kernel.h"
#define SALT 2373
#include "CUDA10_Registers_Kernel.h"
#define SALT 2374
#include "CUDA10_Registers_Kernel.h"
#define SALT 2375
#include "CUDA10_Registers_Kernel.h"
#define SALT 2376
#include "CUDA10_Registers_Kernel.h"
#define SALT 2377
#include "CUDA10_Registers_Kernel.h"
#define SALT 2378
#include "CUDA10_Registers_Kernel.h"
#define SALT 2379
#include "CUDA10_Registers_Kernel.h"
#define SALT 2380
#include "CUDA10_Registers_Kernel.h"
#define SALT 2381
#include "CUDA10_Registers_Kernel.h"
#define SALT 2382
#include "CUDA10_Registers_Kernel.h"
#define SALT 2383
#include "CUDA10_Registers_Kernel.h"
#define SALT 2384
#include "CUDA10_Registers_Kernel.h"
#define SALT 2385
#include "CUDA10_Registers_Kernel.h"
#define SALT 2386
#include "CUDA10_Registers_Kernel.h"
#define SALT 2387
#include "CUDA10_Registers_Kernel.h"
#define SALT 2388
#include "CUDA10_Registers_Kernel.h"
#define SALT 2389
#include "CUDA10_Registers_Kernel.h"
#define SALT 2390
#include "CUDA10_Registers_Kernel.h"
#define SALT 2391
#include "CUDA10_Registers_Kernel.h"
#define SALT 2392
#include "CUDA10_Registers_Kernel.h"
#define SALT 2393
#include "CUDA10_Registers_Kernel.h"
#define SALT 2394
#include "CUDA10_Registers_Kernel.h"
#define SALT 2395
#include "CUDA10_Registers_Kernel.h"
#define SALT 2396
#include "CUDA10_Registers_Kernel.h"
#define SALT 2397
#include "CUDA10_Registers_Kernel.h"
#define SALT 2398
#include "CUDA10_Registers_Kernel.h"
#define SALT 2399
#include "CUDA10_Registers_Kernel.h"
#define SALT 2400
#include "CUDA10_Registers_Kernel.h"
#define SALT 2401
#include "CUDA10_Registers_Kernel.h"
#define SALT 2402
#include "CUDA10_Registers_Kernel.h"
#define SALT 2403
#include "CUDA10_Registers_Kernel.h"
#define SALT 2404
#include "CUDA10_Registers_Kernel.h"
#define SALT 2405
#include "CUDA10_Registers_Kernel.h"
#define SALT 2406
#include "CUDA10_Registers_Kernel.h"
#define SALT 2407
#include "CUDA10_Registers_Kernel.h"
#define SALT 2408
#include "CUDA10_Registers_Kernel.h"
#define SALT 2409
#include "CUDA10_Registers_Kernel.h"
#define SALT 2410
#include "CUDA10_Registers_Kernel.h"
#define SALT 2411
#include "CUDA10_Registers_Kernel.h"
#define SALT 2412
#include "CUDA10_Registers_Kernel.h"
#define SALT 2413
#include "CUDA10_Registers_Kernel.h"
#define SALT 2414
#include "CUDA10_Registers_Kernel.h"
#define SALT 2415
#include "CUDA10_Registers_Kernel.h"
#define SALT 2416
#include "CUDA10_Registers_Kernel.h"
#define SALT 2417
#include "CUDA10_Registers_Kernel.h"
#define SALT 2418
#include "CUDA10_Registers_Kernel.h"
#define SALT 2419
#include "CUDA10_Registers_Kernel.h"
#define SALT 2420
#include "CUDA10_Registers_Kernel.h"
#define SALT 2421
#include "CUDA10_Registers_Kernel.h"
#define SALT 2422
#include "CUDA10_Registers_Kernel.h"
#define SALT 2423
#include "CUDA10_Registers_Kernel.h"
#define SALT 2424
#include "CUDA10_Registers_Kernel.h"
#define SALT 2425
#include "CUDA10_Registers_Kernel.h"
#define SALT 2426
#include "CUDA10_Registers_Kernel.h"
#define SALT 2427
#include "CUDA10_Registers_Kernel.h"
#define SALT 2428
#include "CUDA10_Registers_Kernel.h"
#define SALT 2429
#include "CUDA10_Registers_Kernel.h"
#define SALT 2430
#include "CUDA10_Registers_Kernel.h"
#define SALT 2431
#include "CUDA10_Registers_Kernel.h"
#define SALT 2432
#include "CUDA10_Registers_Kernel.h"
#define SALT 2433
#include "CUDA10_Registers_Kernel.h"
#define SALT 2434
#include "CUDA10_Registers_Kernel.h"
#define SALT 2435
#include "CUDA10_Registers_Kernel.h"
#define SALT 2436
#include "CUDA10_Registers_Kernel.h"
#define SALT 2437
#include "CUDA10_Registers_Kernel.h"
#define SALT 2438
#include "CUDA10_Registers_Kernel.h"
#define SALT 2439
#include "CUDA10_Registers_Kernel.h"
#define SALT 2440
#include "CUDA10_Registers_Kernel.h"
#define SALT 2441
#include "CUDA10_Registers_Kernel.h"
#define SALT 2442
#include "CUDA10_Registers_Kernel.h"
#define SALT 2443
#include "CUDA10_Registers_Kernel.h"
#define SALT 2444
#include "CUDA10_Registers_Kernel.h"
#define SALT 2445
#include "CUDA10_Registers_Kernel.h"
#define SALT 2446
#include "CUDA10_Registers_Kernel.h"
#define SALT 2447
#include "CUDA10_Registers_Kernel.h"
#define SALT 2448
#include "CUDA10_Registers_Kernel.h"
#define SALT 2449
#include "CUDA10_Registers_Kernel.h"
#define SALT 2450
#include "CUDA10_Registers_Kernel.h"
#define SALT 2451
#include "CUDA10_Registers_Kernel.h"
#define SALT 2452
#include "CUDA10_Registers_Kernel.h"
#define SALT 2453
#include "CUDA10_Registers_Kernel.h"
#define SALT 2454
#include "CUDA10_Registers_Kernel.h"
#define SALT 2455
#include "CUDA10_Registers_Kernel.h"
#define SALT 2456
#include "CUDA10_Registers_Kernel.h"
#define SALT 2457
#include "CUDA10_Registers_Kernel.h"
#define SALT 2458
#include "CUDA10_Registers_Kernel.h"
#define SALT 2459
#include "CUDA10_Registers_Kernel.h"
#define SALT 2460
#include "CUDA10_Registers_Kernel.h"
#define SALT 2461
#include "CUDA10_Registers_Kernel.h"
#define SALT 2462
#include "CUDA10_Registers_Kernel.h"
#define SALT 2463
#include "CUDA10_Registers_Kernel.h"
#define SALT 2464
#include "CUDA10_Registers_Kernel.h"
#define SALT 2465
#include "CUDA10_Registers_Kernel.h"
#define SALT 2466
#include "CUDA10_Registers_Kernel.h"
#define SALT 2467
#include "CUDA10_Registers_Kernel.h"
#define SALT 2468
#include "CUDA10_Registers_Kernel.h"
#define SALT 2469
#include "CUDA10_Registers_Kernel.h"
#define SALT 2470
#include "CUDA10_Registers_Kernel.h"
#define SALT 2471
#include "CUDA10_Registers_Kernel.h"
#define SALT 2472
#include "CUDA10_Registers_Kernel.h"
#define SALT 2473
#include "CUDA10_Registers_Kernel.h"
#define SALT 2474
#include "CUDA10_Registers_Kernel.h"
#define SALT 2475
#include "CUDA10_Registers_Kernel.h"
#define SALT 2476
#include "CUDA10_Registers_Kernel.h"
#define SALT 2477
#include "CUDA10_Registers_Kernel.h"
#define SALT 2478
#include "CUDA10_Registers_Kernel.h"
#define SALT 2479
#include "CUDA10_Registers_Kernel.h"
#define SALT 2480
#include "CUDA10_Registers_Kernel.h"
#define SALT 2481
#include "CUDA10_Registers_Kernel.h"
#define SALT 2482
#include "CUDA10_Registers_Kernel.h"
#define SALT 2483
#include "CUDA10_Registers_Kernel.h"
#define SALT 2484
#include "CUDA10_Registers_Kernel.h"
#define SALT 2485
#include "CUDA10_Registers_Kernel.h"
#define SALT 2486
#include "CUDA10_Registers_Kernel.h"
#define SALT 2487
#include "CUDA10_Registers_Kernel.h"
#define SALT 2488
#include "CUDA10_Registers_Kernel.h"
#define SALT 2489
#include "CUDA10_Registers_Kernel.h"
#define SALT 2490
#include "CUDA10_Registers_Kernel.h"
#define SALT 2491
#include "CUDA10_Registers_Kernel.h"
#define SALT 2492
#include "CUDA10_Registers_Kernel.h"
#define SALT 2493
#include "CUDA10_Registers_Kernel.h"
#define SALT 2494
#include "CUDA10_Registers_Kernel.h"
#define SALT 2495
#include "CUDA10_Registers_Kernel.h"
#define SALT 2496
#include "CUDA10_Registers_Kernel.h"
#define SALT 2497
#include "CUDA10_Registers_Kernel.h"
#define SALT 2498
#include "CUDA10_Registers_Kernel.h"
#define SALT 2499
#include "CUDA10_Registers_Kernel.h"
#define SALT 2500
#include "CUDA10_Registers_Kernel.h"
#define SALT 2501
#include "CUDA10_Registers_Kernel.h"
#define SALT 2502
#include "CUDA10_Registers_Kernel.h"
#define SALT 2503
#include "CUDA10_Registers_Kernel.h"
#define SALT 2504
#include "CUDA10_Registers_Kernel.h"
#define SALT 2505
#include "CUDA10_Registers_Kernel.h"
#define SALT 2506
#include "CUDA10_Registers_Kernel.h"
#define SALT 2507
#include "CUDA10_Registers_Kernel.h"
#define SALT 2508
#include "CUDA10_Registers_Kernel.h"
#define SALT 2509
#include "CUDA10_Registers_Kernel.h"
#define SALT 2510
#include "CUDA10_Registers_Kernel.h"
#define SALT 2511
#include "CUDA10_Registers_Kernel.h"
#define SALT 2512
#include "CUDA10_Registers_Kernel.h"
#define SALT 2513
#include "CUDA10_Registers_Kernel.h"
#define SALT 2514
#include "CUDA10_Registers_Kernel.h"
#define SALT 2515
#include "CUDA10_Registers_Kernel.h"
#define SALT 2516
#include "CUDA10_Registers_Kernel.h"
#define SALT 2517
#include "CUDA10_Registers_Kernel.h"
#define SALT 2518
#include "CUDA10_Registers_Kernel.h"
#define SALT 2519
#include "CUDA10_Registers_Kernel.h"
#define SALT 2520
#include "CUDA10_Registers_Kernel.h"
#define SALT 2521
#include "CUDA10_Registers_Kernel.h"
#define SALT 2522
#include "CUDA10_Registers_Kernel.h"
#define SALT 2523
#include "CUDA10_Registers_Kernel.h"
#define SALT 2524
#include "CUDA10_Registers_Kernel.h"
#define SALT 2525
#include "CUDA10_Registers_Kernel.h"
#define SALT 2526
#include "CUDA10_Registers_Kernel.h"
#define SALT 2527
#include "CUDA10_Registers_Kernel.h"
#define SALT 2528
#include "CUDA10_Registers_Kernel.h"
#define SALT 2529
#include "CUDA10_Registers_Kernel.h"
#define SALT 2530
#include "CUDA10_Registers_Kernel.h"
#define SALT 2531
#include "CUDA10_Registers_Kernel.h"
#define SALT 2532
#include "CUDA10_Registers_Kernel.h"
#define SALT 2533
#include "CUDA10_Registers_Kernel.h"
#define SALT 2534
#include "CUDA10_Registers_Kernel.h"
#define SALT 2535
#include "CUDA10_Registers_Kernel.h"
#define SALT 2536
#include "CUDA10_Registers_Kernel.h"
#define SALT 2537
#include "CUDA10_Registers_Kernel.h"
#define SALT 2538
#include "CUDA10_Registers_Kernel.h"
#define SALT 2539
#include "CUDA10_Registers_Kernel.h"
#define SALT 2540
#include "CUDA10_Registers_Kernel.h"
#define SALT 2541
#include "CUDA10_Registers_Kernel.h"
#define SALT 2542
#include "CUDA10_Registers_Kernel.h"
#define SALT 2543
#include "CUDA10_Registers_Kernel.h"
#define SALT 2544
#include "CUDA10_Registers_Kernel.h"
#define SALT 2545
#include "CUDA10_Registers_Kernel.h"
#define SALT 2546
#include "CUDA10_Registers_Kernel.h"
#define SALT 2547
#include "CUDA10_Registers_Kernel.h"
#define SALT 2548
#include "CUDA10_Registers_Kernel.h"
#define SALT 2549
#include "CUDA10_Registers_Kernel.h"
#define SALT 2550
#include "CUDA10_Registers_Kernel.h"
#define SALT 2551
#include "CUDA10_Registers_Kernel.h"
#define SALT 2552
#include "CUDA10_Registers_Kernel.h"
#define SALT 2553
#include "CUDA10_Registers_Kernel.h"
#define SALT 2554
#include "CUDA10_Registers_Kernel.h"
#define SALT 2555
#include "CUDA10_Registers_Kernel.h"
#define SALT 2556
#include "CUDA10_Registers_Kernel.h"
#define SALT 2557
#include "CUDA10_Registers_Kernel.h"
#define SALT 2558
#include "CUDA10_Registers_Kernel.h"
#define SALT 2559
#include "CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher4()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel4(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 2048: LAUNCH_KERNEL(2048); break;
	case 2049: LAUNCH_KERNEL(2049); break;
	case 2050: LAUNCH_KERNEL(2050); break;
	case 2051: LAUNCH_KERNEL(2051); break;
	case 2052: LAUNCH_KERNEL(2052); break;
	case 2053: LAUNCH_KERNEL(2053); break;
	case 2054: LAUNCH_KERNEL(2054); break;
	case 2055: LAUNCH_KERNEL(2055); break;
	case 2056: LAUNCH_KERNEL(2056); break;
	case 2057: LAUNCH_KERNEL(2057); break;
	case 2058: LAUNCH_KERNEL(2058); break;
	case 2059: LAUNCH_KERNEL(2059); break;
	case 2060: LAUNCH_KERNEL(2060); break;
	case 2061: LAUNCH_KERNEL(2061); break;
	case 2062: LAUNCH_KERNEL(2062); break;
	case 2063: LAUNCH_KERNEL(2063); break;
	case 2064: LAUNCH_KERNEL(2064); break;
	case 2065: LAUNCH_KERNEL(2065); break;
	case 2066: LAUNCH_KERNEL(2066); break;
	case 2067: LAUNCH_KERNEL(2067); break;
	case 2068: LAUNCH_KERNEL(2068); break;
	case 2069: LAUNCH_KERNEL(2069); break;
	case 2070: LAUNCH_KERNEL(2070); break;
	case 2071: LAUNCH_KERNEL(2071); break;
	case 2072: LAUNCH_KERNEL(2072); break;
	case 2073: LAUNCH_KERNEL(2073); break;
	case 2074: LAUNCH_KERNEL(2074); break;
	case 2075: LAUNCH_KERNEL(2075); break;
	case 2076: LAUNCH_KERNEL(2076); break;
	case 2077: LAUNCH_KERNEL(2077); break;
	case 2078: LAUNCH_KERNEL(2078); break;
	case 2079: LAUNCH_KERNEL(2079); break;
	case 2080: LAUNCH_KERNEL(2080); break;
	case 2081: LAUNCH_KERNEL(2081); break;
	case 2082: LAUNCH_KERNEL(2082); break;
	case 2083: LAUNCH_KERNEL(2083); break;
	case 2084: LAUNCH_KERNEL(2084); break;
	case 2085: LAUNCH_KERNEL(2085); break;
	case 2086: LAUNCH_KERNEL(2086); break;
	case 2087: LAUNCH_KERNEL(2087); break;
	case 2088: LAUNCH_KERNEL(2088); break;
	case 2089: LAUNCH_KERNEL(2089); break;
	case 2090: LAUNCH_KERNEL(2090); break;
	case 2091: LAUNCH_KERNEL(2091); break;
	case 2092: LAUNCH_KERNEL(2092); break;
	case 2093: LAUNCH_KERNEL(2093); break;
	case 2094: LAUNCH_KERNEL(2094); break;
	case 2095: LAUNCH_KERNEL(2095); break;
	case 2096: LAUNCH_KERNEL(2096); break;
	case 2097: LAUNCH_KERNEL(2097); break;
	case 2098: LAUNCH_KERNEL(2098); break;
	case 2099: LAUNCH_KERNEL(2099); break;
	case 2100: LAUNCH_KERNEL(2100); break;
	case 2101: LAUNCH_KERNEL(2101); break;
	case 2102: LAUNCH_KERNEL(2102); break;
	case 2103: LAUNCH_KERNEL(2103); break;
	case 2104: LAUNCH_KERNEL(2104); break;
	case 2105: LAUNCH_KERNEL(2105); break;
	case 2106: LAUNCH_KERNEL(2106); break;
	case 2107: LAUNCH_KERNEL(2107); break;
	case 2108: LAUNCH_KERNEL(2108); break;
	case 2109: LAUNCH_KERNEL(2109); break;
	case 2110: LAUNCH_KERNEL(2110); break;
	case 2111: LAUNCH_KERNEL(2111); break;
	case 2112: LAUNCH_KERNEL(2112); break;
	case 2113: LAUNCH_KERNEL(2113); break;
	case 2114: LAUNCH_KERNEL(2114); break;
	case 2115: LAUNCH_KERNEL(2115); break;
	case 2116: LAUNCH_KERNEL(2116); break;
	case 2117: LAUNCH_KERNEL(2117); break;
	case 2118: LAUNCH_KERNEL(2118); break;
	case 2119: LAUNCH_KERNEL(2119); break;
	case 2120: LAUNCH_KERNEL(2120); break;
	case 2121: LAUNCH_KERNEL(2121); break;
	case 2122: LAUNCH_KERNEL(2122); break;
	case 2123: LAUNCH_KERNEL(2123); break;
	case 2124: LAUNCH_KERNEL(2124); break;
	case 2125: LAUNCH_KERNEL(2125); break;
	case 2126: LAUNCH_KERNEL(2126); break;
	case 2127: LAUNCH_KERNEL(2127); break;
	case 2128: LAUNCH_KERNEL(2128); break;
	case 2129: LAUNCH_KERNEL(2129); break;
	case 2130: LAUNCH_KERNEL(2130); break;
	case 2131: LAUNCH_KERNEL(2131); break;
	case 2132: LAUNCH_KERNEL(2132); break;
	case 2133: LAUNCH_KERNEL(2133); break;
	case 2134: LAUNCH_KERNEL(2134); break;
	case 2135: LAUNCH_KERNEL(2135); break;
	case 2136: LAUNCH_KERNEL(2136); break;
	case 2137: LAUNCH_KERNEL(2137); break;
	case 2138: LAUNCH_KERNEL(2138); break;
	case 2139: LAUNCH_KERNEL(2139); break;
	case 2140: LAUNCH_KERNEL(2140); break;
	case 2141: LAUNCH_KERNEL(2141); break;
	case 2142: LAUNCH_KERNEL(2142); break;
	case 2143: LAUNCH_KERNEL(2143); break;
	case 2144: LAUNCH_KERNEL(2144); break;
	case 2145: LAUNCH_KERNEL(2145); break;
	case 2146: LAUNCH_KERNEL(2146); break;
	case 2147: LAUNCH_KERNEL(2147); break;
	case 2148: LAUNCH_KERNEL(2148); break;
	case 2149: LAUNCH_KERNEL(2149); break;
	case 2150: LAUNCH_KERNEL(2150); break;
	case 2151: LAUNCH_KERNEL(2151); break;
	case 2152: LAUNCH_KERNEL(2152); break;
	case 2153: LAUNCH_KERNEL(2153); break;
	case 2154: LAUNCH_KERNEL(2154); break;
	case 2155: LAUNCH_KERNEL(2155); break;
	case 2156: LAUNCH_KERNEL(2156); break;
	case 2157: LAUNCH_KERNEL(2157); break;
	case 2158: LAUNCH_KERNEL(2158); break;
	case 2159: LAUNCH_KERNEL(2159); break;
	case 2160: LAUNCH_KERNEL(2160); break;
	case 2161: LAUNCH_KERNEL(2161); break;
	case 2162: LAUNCH_KERNEL(2162); break;
	case 2163: LAUNCH_KERNEL(2163); break;
	case 2164: LAUNCH_KERNEL(2164); break;
	case 2165: LAUNCH_KERNEL(2165); break;
	case 2166: LAUNCH_KERNEL(2166); break;
	case 2167: LAUNCH_KERNEL(2167); break;
	case 2168: LAUNCH_KERNEL(2168); break;
	case 2169: LAUNCH_KERNEL(2169); break;
	case 2170: LAUNCH_KERNEL(2170); break;
	case 2171: LAUNCH_KERNEL(2171); break;
	case 2172: LAUNCH_KERNEL(2172); break;
	case 2173: LAUNCH_KERNEL(2173); break;
	case 2174: LAUNCH_KERNEL(2174); break;
	case 2175: LAUNCH_KERNEL(2175); break;
	case 2176: LAUNCH_KERNEL(2176); break;
	case 2177: LAUNCH_KERNEL(2177); break;
	case 2178: LAUNCH_KERNEL(2178); break;
	case 2179: LAUNCH_KERNEL(2179); break;
	case 2180: LAUNCH_KERNEL(2180); break;
	case 2181: LAUNCH_KERNEL(2181); break;
	case 2182: LAUNCH_KERNEL(2182); break;
	case 2183: LAUNCH_KERNEL(2183); break;
	case 2184: LAUNCH_KERNEL(2184); break;
	case 2185: LAUNCH_KERNEL(2185); break;
	case 2186: LAUNCH_KERNEL(2186); break;
	case 2187: LAUNCH_KERNEL(2187); break;
	case 2188: LAUNCH_KERNEL(2188); break;
	case 2189: LAUNCH_KERNEL(2189); break;
	case 2190: LAUNCH_KERNEL(2190); break;
	case 2191: LAUNCH_KERNEL(2191); break;
	case 2192: LAUNCH_KERNEL(2192); break;
	case 2193: LAUNCH_KERNEL(2193); break;
	case 2194: LAUNCH_KERNEL(2194); break;
	case 2195: LAUNCH_KERNEL(2195); break;
	case 2196: LAUNCH_KERNEL(2196); break;
	case 2197: LAUNCH_KERNEL(2197); break;
	case 2198: LAUNCH_KERNEL(2198); break;
	case 2199: LAUNCH_KERNEL(2199); break;
	case 2200: LAUNCH_KERNEL(2200); break;
	case 2201: LAUNCH_KERNEL(2201); break;
	case 2202: LAUNCH_KERNEL(2202); break;
	case 2203: LAUNCH_KERNEL(2203); break;
	case 2204: LAUNCH_KERNEL(2204); break;
	case 2205: LAUNCH_KERNEL(2205); break;
	case 2206: LAUNCH_KERNEL(2206); break;
	case 2207: LAUNCH_KERNEL(2207); break;
	case 2208: LAUNCH_KERNEL(2208); break;
	case 2209: LAUNCH_KERNEL(2209); break;
	case 2210: LAUNCH_KERNEL(2210); break;
	case 2211: LAUNCH_KERNEL(2211); break;
	case 2212: LAUNCH_KERNEL(2212); break;
	case 2213: LAUNCH_KERNEL(2213); break;
	case 2214: LAUNCH_KERNEL(2214); break;
	case 2215: LAUNCH_KERNEL(2215); break;
	case 2216: LAUNCH_KERNEL(2216); break;
	case 2217: LAUNCH_KERNEL(2217); break;
	case 2218: LAUNCH_KERNEL(2218); break;
	case 2219: LAUNCH_KERNEL(2219); break;
	case 2220: LAUNCH_KERNEL(2220); break;
	case 2221: LAUNCH_KERNEL(2221); break;
	case 2222: LAUNCH_KERNEL(2222); break;
	case 2223: LAUNCH_KERNEL(2223); break;
	case 2224: LAUNCH_KERNEL(2224); break;
	case 2225: LAUNCH_KERNEL(2225); break;
	case 2226: LAUNCH_KERNEL(2226); break;
	case 2227: LAUNCH_KERNEL(2227); break;
	case 2228: LAUNCH_KERNEL(2228); break;
	case 2229: LAUNCH_KERNEL(2229); break;
	case 2230: LAUNCH_KERNEL(2230); break;
	case 2231: LAUNCH_KERNEL(2231); break;
	case 2232: LAUNCH_KERNEL(2232); break;
	case 2233: LAUNCH_KERNEL(2233); break;
	case 2234: LAUNCH_KERNEL(2234); break;
	case 2235: LAUNCH_KERNEL(2235); break;
	case 2236: LAUNCH_KERNEL(2236); break;
	case 2237: LAUNCH_KERNEL(2237); break;
	case 2238: LAUNCH_KERNEL(2238); break;
	case 2239: LAUNCH_KERNEL(2239); break;
	case 2240: LAUNCH_KERNEL(2240); break;
	case 2241: LAUNCH_KERNEL(2241); break;
	case 2242: LAUNCH_KERNEL(2242); break;
	case 2243: LAUNCH_KERNEL(2243); break;
	case 2244: LAUNCH_KERNEL(2244); break;
	case 2245: LAUNCH_KERNEL(2245); break;
	case 2246: LAUNCH_KERNEL(2246); break;
	case 2247: LAUNCH_KERNEL(2247); break;
	case 2248: LAUNCH_KERNEL(2248); break;
	case 2249: LAUNCH_KERNEL(2249); break;
	case 2250: LAUNCH_KERNEL(2250); break;
	case 2251: LAUNCH_KERNEL(2251); break;
	case 2252: LAUNCH_KERNEL(2252); break;
	case 2253: LAUNCH_KERNEL(2253); break;
	case 2254: LAUNCH_KERNEL(2254); break;
	case 2255: LAUNCH_KERNEL(2255); break;
	case 2256: LAUNCH_KERNEL(2256); break;
	case 2257: LAUNCH_KERNEL(2257); break;
	case 2258: LAUNCH_KERNEL(2258); break;
	case 2259: LAUNCH_KERNEL(2259); break;
	case 2260: LAUNCH_KERNEL(2260); break;
	case 2261: LAUNCH_KERNEL(2261); break;
	case 2262: LAUNCH_KERNEL(2262); break;
	case 2263: LAUNCH_KERNEL(2263); break;
	case 2264: LAUNCH_KERNEL(2264); break;
	case 2265: LAUNCH_KERNEL(2265); break;
	case 2266: LAUNCH_KERNEL(2266); break;
	case 2267: LAUNCH_KERNEL(2267); break;
	case 2268: LAUNCH_KERNEL(2268); break;
	case 2269: LAUNCH_KERNEL(2269); break;
	case 2270: LAUNCH_KERNEL(2270); break;
	case 2271: LAUNCH_KERNEL(2271); break;
	case 2272: LAUNCH_KERNEL(2272); break;
	case 2273: LAUNCH_KERNEL(2273); break;
	case 2274: LAUNCH_KERNEL(2274); break;
	case 2275: LAUNCH_KERNEL(2275); break;
	case 2276: LAUNCH_KERNEL(2276); break;
	case 2277: LAUNCH_KERNEL(2277); break;
	case 2278: LAUNCH_KERNEL(2278); break;
	case 2279: LAUNCH_KERNEL(2279); break;
	case 2280: LAUNCH_KERNEL(2280); break;
	case 2281: LAUNCH_KERNEL(2281); break;
	case 2282: LAUNCH_KERNEL(2282); break;
	case 2283: LAUNCH_KERNEL(2283); break;
	case 2284: LAUNCH_KERNEL(2284); break;
	case 2285: LAUNCH_KERNEL(2285); break;
	case 2286: LAUNCH_KERNEL(2286); break;
	case 2287: LAUNCH_KERNEL(2287); break;
	case 2288: LAUNCH_KERNEL(2288); break;
	case 2289: LAUNCH_KERNEL(2289); break;
	case 2290: LAUNCH_KERNEL(2290); break;
	case 2291: LAUNCH_KERNEL(2291); break;
	case 2292: LAUNCH_KERNEL(2292); break;
	case 2293: LAUNCH_KERNEL(2293); break;
	case 2294: LAUNCH_KERNEL(2294); break;
	case 2295: LAUNCH_KERNEL(2295); break;
	case 2296: LAUNCH_KERNEL(2296); break;
	case 2297: LAUNCH_KERNEL(2297); break;
	case 2298: LAUNCH_KERNEL(2298); break;
	case 2299: LAUNCH_KERNEL(2299); break;
	case 2300: LAUNCH_KERNEL(2300); break;
	case 2301: LAUNCH_KERNEL(2301); break;
	case 2302: LAUNCH_KERNEL(2302); break;
	case 2303: LAUNCH_KERNEL(2303); break;
	case 2304: LAUNCH_KERNEL(2304); break;
	case 2305: LAUNCH_KERNEL(2305); break;
	case 2306: LAUNCH_KERNEL(2306); break;
	case 2307: LAUNCH_KERNEL(2307); break;
	case 2308: LAUNCH_KERNEL(2308); break;
	case 2309: LAUNCH_KERNEL(2309); break;
	case 2310: LAUNCH_KERNEL(2310); break;
	case 2311: LAUNCH_KERNEL(2311); break;
	case 2312: LAUNCH_KERNEL(2312); break;
	case 2313: LAUNCH_KERNEL(2313); break;
	case 2314: LAUNCH_KERNEL(2314); break;
	case 2315: LAUNCH_KERNEL(2315); break;
	case 2316: LAUNCH_KERNEL(2316); break;
	case 2317: LAUNCH_KERNEL(2317); break;
	case 2318: LAUNCH_KERNEL(2318); break;
	case 2319: LAUNCH_KERNEL(2319); break;
	case 2320: LAUNCH_KERNEL(2320); break;
	case 2321: LAUNCH_KERNEL(2321); break;
	case 2322: LAUNCH_KERNEL(2322); break;
	case 2323: LAUNCH_KERNEL(2323); break;
	case 2324: LAUNCH_KERNEL(2324); break;
	case 2325: LAUNCH_KERNEL(2325); break;
	case 2326: LAUNCH_KERNEL(2326); break;
	case 2327: LAUNCH_KERNEL(2327); break;
	case 2328: LAUNCH_KERNEL(2328); break;
	case 2329: LAUNCH_KERNEL(2329); break;
	case 2330: LAUNCH_KERNEL(2330); break;
	case 2331: LAUNCH_KERNEL(2331); break;
	case 2332: LAUNCH_KERNEL(2332); break;
	case 2333: LAUNCH_KERNEL(2333); break;
	case 2334: LAUNCH_KERNEL(2334); break;
	case 2335: LAUNCH_KERNEL(2335); break;
	case 2336: LAUNCH_KERNEL(2336); break;
	case 2337: LAUNCH_KERNEL(2337); break;
	case 2338: LAUNCH_KERNEL(2338); break;
	case 2339: LAUNCH_KERNEL(2339); break;
	case 2340: LAUNCH_KERNEL(2340); break;
	case 2341: LAUNCH_KERNEL(2341); break;
	case 2342: LAUNCH_KERNEL(2342); break;
	case 2343: LAUNCH_KERNEL(2343); break;
	case 2344: LAUNCH_KERNEL(2344); break;
	case 2345: LAUNCH_KERNEL(2345); break;
	case 2346: LAUNCH_KERNEL(2346); break;
	case 2347: LAUNCH_KERNEL(2347); break;
	case 2348: LAUNCH_KERNEL(2348); break;
	case 2349: LAUNCH_KERNEL(2349); break;
	case 2350: LAUNCH_KERNEL(2350); break;
	case 2351: LAUNCH_KERNEL(2351); break;
	case 2352: LAUNCH_KERNEL(2352); break;
	case 2353: LAUNCH_KERNEL(2353); break;
	case 2354: LAUNCH_KERNEL(2354); break;
	case 2355: LAUNCH_KERNEL(2355); break;
	case 2356: LAUNCH_KERNEL(2356); break;
	case 2357: LAUNCH_KERNEL(2357); break;
	case 2358: LAUNCH_KERNEL(2358); break;
	case 2359: LAUNCH_KERNEL(2359); break;
	case 2360: LAUNCH_KERNEL(2360); break;
	case 2361: LAUNCH_KERNEL(2361); break;
	case 2362: LAUNCH_KERNEL(2362); break;
	case 2363: LAUNCH_KERNEL(2363); break;
	case 2364: LAUNCH_KERNEL(2364); break;
	case 2365: LAUNCH_KERNEL(2365); break;
	case 2366: LAUNCH_KERNEL(2366); break;
	case 2367: LAUNCH_KERNEL(2367); break;
	case 2368: LAUNCH_KERNEL(2368); break;
	case 2369: LAUNCH_KERNEL(2369); break;
	case 2370: LAUNCH_KERNEL(2370); break;
	case 2371: LAUNCH_KERNEL(2371); break;
	case 2372: LAUNCH_KERNEL(2372); break;
	case 2373: LAUNCH_KERNEL(2373); break;
	case 2374: LAUNCH_KERNEL(2374); break;
	case 2375: LAUNCH_KERNEL(2375); break;
	case 2376: LAUNCH_KERNEL(2376); break;
	case 2377: LAUNCH_KERNEL(2377); break;
	case 2378: LAUNCH_KERNEL(2378); break;
	case 2379: LAUNCH_KERNEL(2379); break;
	case 2380: LAUNCH_KERNEL(2380); break;
	case 2381: LAUNCH_KERNEL(2381); break;
	case 2382: LAUNCH_KERNEL(2382); break;
	case 2383: LAUNCH_KERNEL(2383); break;
	case 2384: LAUNCH_KERNEL(2384); break;
	case 2385: LAUNCH_KERNEL(2385); break;
	case 2386: LAUNCH_KERNEL(2386); break;
	case 2387: LAUNCH_KERNEL(2387); break;
	case 2388: LAUNCH_KERNEL(2388); break;
	case 2389: LAUNCH_KERNEL(2389); break;
	case 2390: LAUNCH_KERNEL(2390); break;
	case 2391: LAUNCH_KERNEL(2391); break;
	case 2392: LAUNCH_KERNEL(2392); break;
	case 2393: LAUNCH_KERNEL(2393); break;
	case 2394: LAUNCH_KERNEL(2394); break;
	case 2395: LAUNCH_KERNEL(2395); break;
	case 2396: LAUNCH_KERNEL(2396); break;
	case 2397: LAUNCH_KERNEL(2397); break;
	case 2398: LAUNCH_KERNEL(2398); break;
	case 2399: LAUNCH_KERNEL(2399); break;
	case 2400: LAUNCH_KERNEL(2400); break;
	case 2401: LAUNCH_KERNEL(2401); break;
	case 2402: LAUNCH_KERNEL(2402); break;
	case 2403: LAUNCH_KERNEL(2403); break;
	case 2404: LAUNCH_KERNEL(2404); break;
	case 2405: LAUNCH_KERNEL(2405); break;
	case 2406: LAUNCH_KERNEL(2406); break;
	case 2407: LAUNCH_KERNEL(2407); break;
	case 2408: LAUNCH_KERNEL(2408); break;
	case 2409: LAUNCH_KERNEL(2409); break;
	case 2410: LAUNCH_KERNEL(2410); break;
	case 2411: LAUNCH_KERNEL(2411); break;
	case 2412: LAUNCH_KERNEL(2412); break;
	case 2413: LAUNCH_KERNEL(2413); break;
	case 2414: LAUNCH_KERNEL(2414); break;
	case 2415: LAUNCH_KERNEL(2415); break;
	case 2416: LAUNCH_KERNEL(2416); break;
	case 2417: LAUNCH_KERNEL(2417); break;
	case 2418: LAUNCH_KERNEL(2418); break;
	case 2419: LAUNCH_KERNEL(2419); break;
	case 2420: LAUNCH_KERNEL(2420); break;
	case 2421: LAUNCH_KERNEL(2421); break;
	case 2422: LAUNCH_KERNEL(2422); break;
	case 2423: LAUNCH_KERNEL(2423); break;
	case 2424: LAUNCH_KERNEL(2424); break;
	case 2425: LAUNCH_KERNEL(2425); break;
	case 2426: LAUNCH_KERNEL(2426); break;
	case 2427: LAUNCH_KERNEL(2427); break;
	case 2428: LAUNCH_KERNEL(2428); break;
	case 2429: LAUNCH_KERNEL(2429); break;
	case 2430: LAUNCH_KERNEL(2430); break;
	case 2431: LAUNCH_KERNEL(2431); break;
	case 2432: LAUNCH_KERNEL(2432); break;
	case 2433: LAUNCH_KERNEL(2433); break;
	case 2434: LAUNCH_KERNEL(2434); break;
	case 2435: LAUNCH_KERNEL(2435); break;
	case 2436: LAUNCH_KERNEL(2436); break;
	case 2437: LAUNCH_KERNEL(2437); break;
	case 2438: LAUNCH_KERNEL(2438); break;
	case 2439: LAUNCH_KERNEL(2439); break;
	case 2440: LAUNCH_KERNEL(2440); break;
	case 2441: LAUNCH_KERNEL(2441); break;
	case 2442: LAUNCH_KERNEL(2442); break;
	case 2443: LAUNCH_KERNEL(2443); break;
	case 2444: LAUNCH_KERNEL(2444); break;
	case 2445: LAUNCH_KERNEL(2445); break;
	case 2446: LAUNCH_KERNEL(2446); break;
	case 2447: LAUNCH_KERNEL(2447); break;
	case 2448: LAUNCH_KERNEL(2448); break;
	case 2449: LAUNCH_KERNEL(2449); break;
	case 2450: LAUNCH_KERNEL(2450); break;
	case 2451: LAUNCH_KERNEL(2451); break;
	case 2452: LAUNCH_KERNEL(2452); break;
	case 2453: LAUNCH_KERNEL(2453); break;
	case 2454: LAUNCH_KERNEL(2454); break;
	case 2455: LAUNCH_KERNEL(2455); break;
	case 2456: LAUNCH_KERNEL(2456); break;
	case 2457: LAUNCH_KERNEL(2457); break;
	case 2458: LAUNCH_KERNEL(2458); break;
	case 2459: LAUNCH_KERNEL(2459); break;
	case 2460: LAUNCH_KERNEL(2460); break;
	case 2461: LAUNCH_KERNEL(2461); break;
	case 2462: LAUNCH_KERNEL(2462); break;
	case 2463: LAUNCH_KERNEL(2463); break;
	case 2464: LAUNCH_KERNEL(2464); break;
	case 2465: LAUNCH_KERNEL(2465); break;
	case 2466: LAUNCH_KERNEL(2466); break;
	case 2467: LAUNCH_KERNEL(2467); break;
	case 2468: LAUNCH_KERNEL(2468); break;
	case 2469: LAUNCH_KERNEL(2469); break;
	case 2470: LAUNCH_KERNEL(2470); break;
	case 2471: LAUNCH_KERNEL(2471); break;
	case 2472: LAUNCH_KERNEL(2472); break;
	case 2473: LAUNCH_KERNEL(2473); break;
	case 2474: LAUNCH_KERNEL(2474); break;
	case 2475: LAUNCH_KERNEL(2475); break;
	case 2476: LAUNCH_KERNEL(2476); break;
	case 2477: LAUNCH_KERNEL(2477); break;
	case 2478: LAUNCH_KERNEL(2478); break;
	case 2479: LAUNCH_KERNEL(2479); break;
	case 2480: LAUNCH_KERNEL(2480); break;
	case 2481: LAUNCH_KERNEL(2481); break;
	case 2482: LAUNCH_KERNEL(2482); break;
	case 2483: LAUNCH_KERNEL(2483); break;
	case 2484: LAUNCH_KERNEL(2484); break;
	case 2485: LAUNCH_KERNEL(2485); break;
	case 2486: LAUNCH_KERNEL(2486); break;
	case 2487: LAUNCH_KERNEL(2487); break;
	case 2488: LAUNCH_KERNEL(2488); break;
	case 2489: LAUNCH_KERNEL(2489); break;
	case 2490: LAUNCH_KERNEL(2490); break;
	case 2491: LAUNCH_KERNEL(2491); break;
	case 2492: LAUNCH_KERNEL(2492); break;
	case 2493: LAUNCH_KERNEL(2493); break;
	case 2494: LAUNCH_KERNEL(2494); break;
	case 2495: LAUNCH_KERNEL(2495); break;
	case 2496: LAUNCH_KERNEL(2496); break;
	case 2497: LAUNCH_KERNEL(2497); break;
	case 2498: LAUNCH_KERNEL(2498); break;
	case 2499: LAUNCH_KERNEL(2499); break;
	case 2500: LAUNCH_KERNEL(2500); break;
	case 2501: LAUNCH_KERNEL(2501); break;
	case 2502: LAUNCH_KERNEL(2502); break;
	case 2503: LAUNCH_KERNEL(2503); break;
	case 2504: LAUNCH_KERNEL(2504); break;
	case 2505: LAUNCH_KERNEL(2505); break;
	case 2506: LAUNCH_KERNEL(2506); break;
	case 2507: LAUNCH_KERNEL(2507); break;
	case 2508: LAUNCH_KERNEL(2508); break;
	case 2509: LAUNCH_KERNEL(2509); break;
	case 2510: LAUNCH_KERNEL(2510); break;
	case 2511: LAUNCH_KERNEL(2511); break;
	case 2512: LAUNCH_KERNEL(2512); break;
	case 2513: LAUNCH_KERNEL(2513); break;
	case 2514: LAUNCH_KERNEL(2514); break;
	case 2515: LAUNCH_KERNEL(2515); break;
	case 2516: LAUNCH_KERNEL(2516); break;
	case 2517: LAUNCH_KERNEL(2517); break;
	case 2518: LAUNCH_KERNEL(2518); break;
	case 2519: LAUNCH_KERNEL(2519); break;
	case 2520: LAUNCH_KERNEL(2520); break;
	case 2521: LAUNCH_KERNEL(2521); break;
	case 2522: LAUNCH_KERNEL(2522); break;
	case 2523: LAUNCH_KERNEL(2523); break;
	case 2524: LAUNCH_KERNEL(2524); break;
	case 2525: LAUNCH_KERNEL(2525); break;
	case 2526: LAUNCH_KERNEL(2526); break;
	case 2527: LAUNCH_KERNEL(2527); break;
	case 2528: LAUNCH_KERNEL(2528); break;
	case 2529: LAUNCH_KERNEL(2529); break;
	case 2530: LAUNCH_KERNEL(2530); break;
	case 2531: LAUNCH_KERNEL(2531); break;
	case 2532: LAUNCH_KERNEL(2532); break;
	case 2533: LAUNCH_KERNEL(2533); break;
	case 2534: LAUNCH_KERNEL(2534); break;
	case 2535: LAUNCH_KERNEL(2535); break;
	case 2536: LAUNCH_KERNEL(2536); break;
	case 2537: LAUNCH_KERNEL(2537); break;
	case 2538: LAUNCH_KERNEL(2538); break;
	case 2539: LAUNCH_KERNEL(2539); break;
	case 2540: LAUNCH_KERNEL(2540); break;
	case 2541: LAUNCH_KERNEL(2541); break;
	case 2542: LAUNCH_KERNEL(2542); break;
	case 2543: LAUNCH_KERNEL(2543); break;
	case 2544: LAUNCH_KERNEL(2544); break;
	case 2545: LAUNCH_KERNEL(2545); break;
	case 2546: LAUNCH_KERNEL(2546); break;
	case 2547: LAUNCH_KERNEL(2547); break;
	case 2548: LAUNCH_KERNEL(2548); break;
	case 2549: LAUNCH_KERNEL(2549); break;
	case 2550: LAUNCH_KERNEL(2550); break;
	case 2551: LAUNCH_KERNEL(2551); break;
	case 2552: LAUNCH_KERNEL(2552); break;
	case 2553: LAUNCH_KERNEL(2553); break;
	case 2554: LAUNCH_KERNEL(2554); break;
	case 2555: LAUNCH_KERNEL(2555); break;
	case 2556: LAUNCH_KERNEL(2556); break;
	case 2557: LAUNCH_KERNEL(2557); break;
	case 2558: LAUNCH_KERNEL(2558); break;
	case 2559: LAUNCH_KERNEL(2559); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
