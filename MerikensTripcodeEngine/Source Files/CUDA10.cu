#include "hip/hip_runtime.h"
// Meriken's Tripcode Engine 1.1 Beta 1
// Copyright (c) 2011-2013 Meriken//XXX <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



// TO DO: Use smallKeyBitmap[]!



///////////////////////////////////////////////////////////////////////////////
// INCLUDE FILE(S)                                                           //
///////////////////////////////////////////////////////////////////////////////

#include "MerikensTripcodeEngine.h"



///////////////////////////////////////////////////////////////////////////////
// VARIABLES FOR CUDA CODES                                                  //
///////////////////////////////////////////////////////////////////////////////

__device__ __constant__ unsigned char   CUDA_keyCharTable_OneByte[SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   CUDA_keyCharTable_FirstByte  [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ unsigned char   CUDA_keyCharTable_SecondByte [SIZE_KEY_CHAR_TABLE];
__device__ __constant__ char    CUDA_base64CharTable[64];
__device__ __constant__ unsigned char   CUDA_key[12];



///////////////////////////////////////////////////////////////////////////////
// BITSLICE DES                                                              //
///////////////////////////////////////////////////////////////////////////////

typedef int           DES_ARCH_WORD;
typedef int           DES_ARCH_WORD_32;
#define DES_ARCH_SIZE 4
#define DES_ARCH_BITS 32

typedef int           DES_Vector;
// #define CUDA_DES_BS_DEPTH  DES_ARCH_BITS
#define DES_VECTOR_ZERO               0
#define DES_VECTOR_ONES               ~(DES_Vector)0

#define DES_VECTOR_NOT(dst, a)        (dst) =  ~(a)
#define DES_VECTOR_AND(dst, a, b)     (dst) =   (a) &  (b)
#define DES_VECTOR_OR(dst, a, b)      (dst) =   (a) |  (b)
#define DES_VECTOR_AND_NOT(dst, a, b) (dst) =   (a) & ~(b)
#define DES_VECTOR_XOR_NOT(dst, a, b) (dst) = ~((a) ^  (b))
#define DES_VECTOR_NOT_OR(dst, a, b)  (dst) = ~((a) |  (b))
#define DES_VECTOR_SEL(dst, a, b, c)  (dst) = (((a) & ~(c)) ^ ((b) & (c)))
#define DES_VECTOR_XOR_FUNC(a, b)              ((a) ^  (b))
#define DES_VECTOR_XOR(dst, a, b)     (dst) = DES_VECTOR_XOR_FUNC((a), (b))
#define DES_VECTOR_SET(dst, ofs, src) *((DES_Vector *)((DES_Vector *)&(dst) + (ofs))) = (src)

#define DES_CONSTANT_QUALIFIERS      __device__ __constant__
#define DES_FUNCTION_QUALIFIERS      __device__ __forceinline__
#define DES_SBOX_FUNCTION_QUALIFIERS __device__ __forceinline__

// All bitslice DES parameters combined into one struct for more efficient
// cache usage and multi-threading.
struct DESContext {
	DES_Vector keys[56];       // 224 bytes
	DES_Vector dataBlocks[64]; // 256 bytes
	unsigned int dummy[1];     // This is necessary to reduce the number of bank conflicts in shared memory.
};
__shared__ struct DESContext DESContextArray[CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK];

const unsigned char expansionTable[48] = {
	31,  0,  1,  2,  3,  4,
	 3,  4,  5,  6,  7,  8,
	 7,  8,  9, 10, 11, 12,
	11, 12, 13, 14, 15, 16,
	15, 16, 17, 18, 19, 20,
	19, 20, 21, 22, 23, 24,
	23, 24, 25, 26, 27, 28,
	27, 28, 29, 30, 31,  0
};

__device__ __constant__ unsigned char CUDA_expansionFunction[96];

const char charToIndexTableForDES[0x100] = {
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x00, 0x01,
	0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09,
	0x0a, 0x0b, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x0c, 0x0d, 0x0e, 0x0f, 0x10, 0x11, 0x12,
	0x13, 0x14, 0x15, 0x16, 0x17, 0x18, 0x19, 0x1a,
	0x1b, 0x1c, 0x1d, 0x1e, 0x1f, 0x20, 0x21, 0x22,
	0x23, 0x24, 0x25, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x26, 0x27, 0x28, 0x29, 0x2a, 0x2b, 0x2c,
	0x2d, 0x2e, 0x2f, 0x30, 0x31, 0x32, 0x33, 0x34,
	0x35, 0x36, 0x37, 0x38, 0x39, 0x3a, 0x3b, 0x3c,
	0x3d, 0x3e, 0x3f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
	0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f, 0x7f,
};

DES_CONSTANT_QUALIFIERS char CUDA_DES_indexToCharTable[64] =
//	"./0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
{
	/* 00 */ '.', '/',
	/* 02 */ '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 
	/* 12 */ 'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 
	/* 28 */ 'Q', 'R', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z',
	/* 38 */ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p',
	/* 54 */ 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z', 
};

DES_CONSTANT_QUALIFIERS unsigned char keySchedule[DES_SIZE_KEY_SCHEDULE] = {
	12, 46, 33, 52, 48, 20, 34, 55,  5, 13, 18, 40,  4, 32, 26, 27,
	38, 54, 53,  6, 31, 25, 19, 41, 15, 24, 28, 43, 30,  3, 35, 22,
	 2, 44, 14, 23, 51, 16, 29, 49,  7, 17, 37,  8,  9, 50, 42, 21,
	 5, 39, 26, 45, 41, 13, 27, 48, 53,  6, 11, 33, 52, 25, 19, 20,
	31, 47, 46, 54, 55, 18, 12, 34,  8, 17, 21, 36, 23, 49, 28, 15,
	24, 37,  7, 16, 44,  9, 22, 42,  0, 10, 30,  1,  2, 43, 35, 14,
	46, 25, 12, 31, 27, 54, 13, 34, 39, 47, 52, 19, 38, 11,  5,  6,
	48, 33, 32, 40, 41,  4, 53, 20, 51,  3,  7, 22,  9, 35, 14,  1,
	10, 23, 50,  2, 30, 24,  8, 28, 43, 49, 16, 44, 17, 29, 21,  0,
	32, 11, 53, 48, 13, 40, 54, 20, 25, 33, 38,  5, 55, 52, 46, 47,
	34, 19, 18, 26, 27, 45, 39,  6, 37, 42, 50,  8, 24, 21,  0, 44,
	49,  9, 36, 17, 16, 10, 51, 14, 29, 35,  2, 30,  3, 15,  7, 43,
	18, 52, 39, 34, 54, 26, 40,  6, 11, 19, 55, 46, 41, 38, 32, 33,
	20,  5,  4, 12, 13, 31, 25, 47, 23, 28, 36, 51, 10,  7, 43, 30,
	35, 24, 22,  3,  2, 49, 37,  0, 15, 21, 17, 16, 42,  1, 50, 29,
	 4, 38, 25, 20, 40, 12, 26, 47, 52,  5, 41, 32, 27, 55, 18, 19,
	 6, 46, 45, 53, 54, 48, 11, 33,  9, 14, 22, 37, 49, 50, 29, 16,
	21, 10,  8, 42, 17, 35, 23, 43,  1,  7,  3,  2, 28, 44, 36, 15,
	45, 55, 11,  6, 26, 53, 12, 33, 38, 46, 27, 18, 13, 41,  4,  5,
	47, 32, 31, 39, 40, 34, 52, 19, 24,  0,  8, 23, 35, 36, 15,  2,
	 7, 49, 51, 28,  3, 21,  9, 29, 44, 50, 42, 17, 14, 30, 22,  1,
	31, 41, 52, 47, 12, 39, 53, 19, 55, 32, 13,  4, 54, 27, 45, 46,
	33, 18, 48, 25, 26, 20, 38,  5, 10, 43, 51,  9, 21, 22,  1, 17,
	50, 35, 37, 14, 42,  7, 24, 15, 30, 36, 28,  3,  0, 16,  8, 44,
	55, 34, 45, 40,  5, 32, 46, 12, 48, 25,  6, 52, 47, 20, 38, 39,
	26, 11, 41, 18, 19, 13, 31, 53,  3, 36, 44,  2, 14, 15, 51, 10,
	43, 28, 30,  7, 35,  0, 17,  8, 23, 29, 21, 49, 50,  9,  1, 37,
	41, 20, 31, 26, 46, 18, 32, 53, 34, 11, 47, 38, 33,  6, 55, 25,
	12, 52, 27,  4,  5, 54, 48, 39, 42, 22, 30, 17,  0,  1, 37, 49,
	29, 14, 16, 50, 21, 43,  3, 51,  9, 15,  7, 35, 36, 24, 44, 23,
	27,  6, 48, 12, 32,  4, 18, 39, 20, 52, 33, 55, 19, 47, 41, 11,
	53, 38, 13, 45, 46, 40, 34, 25, 28,  8, 16,  3, 43, 44, 23, 35,
	15,  0,  2, 36,  7, 29, 42, 37, 24,  1, 50, 21, 22, 10, 30,  9,
	13, 47, 34, 53, 18, 45,  4, 25,  6, 38, 19, 41,  5, 33, 27, 52,
	39, 55, 54, 31, 32, 26, 20, 11, 14, 51,  2, 42, 29, 30,  9, 21,
	 1, 43, 17, 22, 50, 15, 28, 23, 10, 44, 36,  7,  8, 49, 16, 24,
	54, 33, 20, 39,  4, 31, 45, 11, 47, 55,  5, 27, 46, 19, 13, 38,
	25, 41, 40, 48, 18, 12,  6, 52,  0, 37, 17, 28, 15, 16, 24,  7,
	44, 29,  3,  8, 36,  1, 14,  9, 49, 30, 22, 50, 51, 35,  2, 10,
	40, 19,  6, 25, 45, 48, 31, 52, 33, 41, 46, 13, 32,  5, 54, 55,
	11, 27, 26, 34,  4, 53, 47, 38, 43, 23,  3, 14,  1,  2, 10, 50,
	30, 15, 42, 51, 22, 44,  0, 24, 35, 16,  8, 36, 37, 21, 17, 49,
	26,  5, 47, 11, 31, 34, 48, 38, 19, 27, 32, 54, 18, 46, 40, 41,
	52, 13, 12, 20, 45, 39, 33, 55, 29,  9, 42,  0, 44, 17, 49, 36,
	16,  1, 28, 37,  8, 30, 43, 10, 21,  2, 51, 22, 23,  7,  3, 35,
	19, 53, 40,  4, 55, 27, 41, 31, 12, 20, 25, 47, 11, 39, 33, 34,
	45,  6,  5, 13, 38, 32, 26, 48, 22,  2, 35, 50, 37, 10, 42, 29,
	 9, 51, 21, 30,  1, 23, 36,  3, 14, 24, 44, 15, 16,  0, 49, 28,
};

void DES_CreateExpansionFunction(char *saltString, unsigned char *expansionFunction)
{
	unsigned char saltChar1 = '.', saltChar2 = '.';
	DES_ARCH_WORD salt;
	DES_ARCH_WORD mask;
	int src, dst;

	if (saltString[0]) {
		saltChar1 = saltString[0];
		if (saltString[1])
			saltChar2 = saltString[1];
	}
	salt =    charToIndexTableForDES[saltChar1]
	       | (charToIndexTableForDES[saltChar2] << 6);

	mask = 1;
	for (dst = 0; dst < 48; dst++) {
		if (dst == 24) mask = 1;

		if (salt & mask) {
			if (dst < 24) src = dst + 24; else src = dst - 24;
		} else src = dst;

		expansionFunction[dst     ] = expansionTable[src];
		expansionFunction[dst + 48] = expansionTable[src] + 32;

		mask <<= 1;
	}
}

DES_FUNCTION_QUALIFIERS void
DES_ClearKeys(int charIndex)
{
	for (int i = 0; i < 7; ++i)
		DESContextArray[threadIdx.x].keys[charIndex * 7 + i] = 0;
}

// Bitslice DES S-boxes for x86 with MMX/SSE2/AVX and for typical RISC
// architectures.  These use AND, OR, XOR, NOT, and AND-NOT gates.
//
// Gate counts: 49 44 46 33 48 46 46 41
// Average: 44.125
//
// Several same-gate-count expressions for each S-box are included (for use on
// different CPUs/GPUs).
//
// These Boolean expressions corresponding to DES S-boxes have been generated
// by Roman Rusakov <roman_rus at openwall.com> for use in Openwall's
// John the Ripper password cracker: http://www.openwall.com/john/
// Being mathematical formulas, they are not copyrighted and are free for reuse
// by anyone.
//
// This file (a specific representation of the S-box expressions, surrounding
// logic) is Copyright (c) 2011 by Solar Designer <solar at openwall.com>.
// Redistribution and use in source and binary forms, with or without
// modification, are permitted.  (This is a heavily cut-down "BSD license".)
//
// The effort has been sponsored by Rapid7: http://www.rapid7.com

//
// s1-00484, 49 gates, 17 regs, 11 andn, 4/9/39/79/120 stalls, 74 biop
// Currently used for MMX/SSE2 and x86-64 SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s1(
	DES_Vector arg1,
	DES_Vector arg2,
	DES_Vector arg3,
	DES_Vector arg4,
	DES_Vector arg5,
	DES_Vector arg6,
    DES_Vector *out1,
    DES_Vector *out2,
    DES_Vector *out3,
    DES_Vector *out4
) {
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"
	    ".reg .u32 t11;         \n\t"
	    ".reg .u32 t12;         \n\t"
	    ".reg .u32 t13;         \n\t"
	    
	    "not.b32 t0,  %8;      \n\t"
	    "and.b32 t0,  %4, t0;  \n\t"
	    "xor.b32 t1,  %7, t0;  \n\t"
	    "or.b32  t2,  %6, %9; \n\t"
	    "xor.b32 t3,  %4, %6; \n\t"
	    "and.b32 t4,  t2,  t3;  \n\t"
	    "xor.b32 t5,  %7, t4;  \n\t"
	    "not.b32 t6,  t1;       \n\t"
	    "and.b32 t6,  t5,  t6;  \n\t"

	    "xor.b32 t7,  %8, %9; \n\t"
	    "xor.b32 t8,  %6, t7;  \n\t"
	    "not.b32 t9,  t8;       \n\t"
	    "and.b32 t9,  t1,  t9;  \n\t"
	    "or.b32  t8,  %9, t4;  \n\t"
	    "xor.b32 t4,  t9,  t8;  \n\t"
	    "not.b32 t8,  t6;       \n\t"
	    "and.b32 t8,  t4,  t8;  \n\t"

	    "or.b32  t9,  %4, %9; \n\t"
	    "or.b32  t10, t4,  t9;  \n\t"
	    "not.b32 t11, t5;       \n\t"
	    "and.b32 t11, %8, t11; \n\t"
	    "xor.b32 t5,  t10, t11; \n\t"

	    "not.b32 t12, t9;       \n\t"
		"and.b32 t12, %7, t12; \n\t"
	    "xor.b32 t9,  t11, t12; \n\t"
	    "not.b32 t12, t3;       \n\t"
	    "and.b32 t12, t7,  t12; \n\t"
	    "or.b32  t3,  t9,  t12; \n\t"

	    "not.b32 t12, t0;       \n\t"
	    "and.b32 t12, %6, t12; \n\t"
	    "xor.b32 t0,  t1,  t10; \n\t"
	    "not.b32 t9,  t12;      \n\t"
	    "and.b32 t9,  t0,  t9;  \n\t"
	    "not.b32 t12, t9;       \n\t"
	    "and.b32 t0,  t2,  t4;  \n\t"
	    "xor.b32 t4,  t12, t0;  \n\t"
	    "not.b32 t13, %5;      \n\t"
	    "and.b32 t13, t5,  t13; \n\t"
	    "xor.b32 t5,  t13, t4;  \n\t"
	    "xor.b32 %2, %2, t5;  \n\t"
	
	    "xor.b32 t12, t7,  t9;  \n\t"
	    "or.b32  t0,  t11, t12; \n\t"
	    "xor.b32 t5,  t2,  t0;  \n\t"
	    "xor.b32 t11, %4, t5;  \n\t"
	    "xor.b32 t5,  t4,  t11; \n\t"
	    "or.b32  t9,  t6,  %5; \n\t"
	    "xor.b32 t12, t9,  t5;  \n\t"
	    "xor.b32 %0, %0, t12; \n\t"
	
	    "xor.b32 t13, t2,  t10; \n\t"
	    "or.b32  t0,  t3,  t13; \n\t"
	    "xor.b32 t13, t11, t0;  \n\t"
	    "or.b32  t0,  t7,  t5;  \n\t"
	    "xor.b32 t5,  t13, t0;  \n\t"
	    "or.b32  t0,  t8,  %5; \n\t"
	    "xor.b32 t6,  t0,  t5;  \n\t"
	    "xor.b32 %1, %1, t6;  \n\t"

	    "or.b32  t6,  %8, t1;  \n\t"
	    "not.b32 t9,  t13;      \n\t"
	    "and.b32 t9,  t6,  t9;  \n\t"
	    "and.b32 t13, t8,  t11; \n\t"
	    "xor.b32 t11, t9,  t13; \n\t"
	    "or.b32  t13, t11, %5; \n\t"
	    "xor.b32 t12, t13, t3;  \n\t"
	    "xor.b32 %3, %3, t12;   \n\t"
	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(arg1)     // %4
	      "r"(arg2)     // %5
	      "r"(arg3)     // %6
	      "r"(arg4)     // %7
	      "r"(arg5)     // %8
	      "r"(arg6));   // %9
}

//
// s2-016251, 44 gates, 14 regs, 13 andn, 1/9/22/61/108 stalls, 66 biop */
//
DES_SBOX_FUNCTION_QUALIFIERS void
s2(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"
	    ".reg .u32 t11;         \n\t"
	    ".reg .u32 t12;         \n\t"

		"xor.b32 t0, %5, %8;    \n\t"

		"not.b32 t1, %9;        \n\t"
		"and.b32 t1, %4, t1;    \n\t"
		"not.b32 t2, t1;        \n\t"
		"and.b32 t2, %8, t2;    \n\t"
		"or.b32  t1, %5, t2;    \n\t"

		"not.b32 t3, %9;        \n\t"
		"and.b32 t3, t0, t3;    \n\t"
		"and.b32 t4, %4, t0;    \n\t"
		"xor.b32 t5, %8, t4;    \n\t"
		"not.b32 t6, t3;        \n\t"
		"and.b32 t6, t5, t6;    \n\t"

		"and.b32 t7, %6, %9;    \n\t"
		"xor.b32 t8, t2, t3;    \n\t"
		"and.b32 t2, t1, t8;    \n\t"
		"not.b32 t3, t7;        \n\t"
		"and.b32 t3, t2, t3;    \n\t"

		"and.b32 t8, %6, t2;    \n\t"
		"not.b32 t2, %4;        \n\t"
		"xor.b32 t9, t8, t2;    \n\t"
		"xor.b32 t2, %9, t0;    \n\t"
		"not.b32 t0, t7;        \n\t"
		"and.b32 t0, t2, t0;    \n\t"
		"xor.b32 t10, t9, t0;   \n\t"
		"not.b32 t11, t3;       \n\t"
		"and.b32 t11, %7, t11;  \n\t"
		"xor.b32 t3, t11, t10;  \n\t"
		"xor.b32 %1, %1, t3;    \n\t"

		"not.b32 t3, t0;        \n\t"
		"and.b32 t3, %5, t3;    \n\t"
		"xor.b32 t0, t5, t3;    \n\t"
		"not.b32 t5, t0;        \n\t"
		"and.b32 t5, t9, t5;    \n\t"
		"xor.b32 t9, %6, t2;    \n\t"
		"xor.b32 t11, t5, t9;   \n\t"
		"not.b32 t5, %7;        \n\t"
		"and.b32 t5, t1, t5;    \n\t"
		"xor.b32 t12, t5, t11;  \n\t"
		"xor.b32 %0, %0, t12;   \n\t"

		"xor.b32 t5, t8, t3;    \n\t"
		"or.b32  t3, t9, t5;    \n\t"
		"xor.b32 t8, t1, t10;   \n\t"
		"or.b32  t1, t7, t8;    \n\t"
		"xor.b32 t7, t3, t1;    \n\t"

		"not.b32 t1, t11;       \n\t"
		"and.b32 t1, t10, t1;   \n\t"
		"xor.b32 t3, t4, t5;    \n\t"
		"or.b32  t4, t1, t3;    \n\t"
		"not.b32 t1, t9;        \n\t"
		"and.b32 t1, t6, t1;    \n\t"
		"xor.b32 t3, t4, t1;    \n\t"
		"or.b32  t1, t3, %7;    \n\t"
		"xor.b32 t4, t1, t7;    \n\t"
		"xor.b32 %2, %2, t4;    \n\t"

		"not.b32 t1, t0;        \n\t"
		"and.b32 t1, t3, t1;    \n\t"
		"or.b32  t0, t2, t8;    \n\t"
		"xor.b32 t2, t1, t0;    \n\t"
		"or.b32  t0, t6, %7;    \n\t"
		"xor.b32 t1, t0, t2;    \n\t"
		"xor.b32 %3, %3, t1;    \n\t"
		
		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s3-000426, 46 gates, 16 regs, 14 andn, 2/5/12/35/75 stalls, 68 biop
// Currently used for x86-64 SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s3(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"

		"not.b32 t0, %5;        \n\t"
		"and.b32 t0, %4, t0;    \n\t"
		"xor.b32 t1, %6, %9;    \n\t"
		"or.b32  t2, t0, t1;    \n\t"
		"xor.b32 t0, %7, %9;    \n\t"
		"not.b32 t3, %4;        \n\t"
		"and.b32 t3, t0, t3;    \n\t"
		"xor.b32 t4, t2, t3;    \n\t"

		"xor.b32 t5, %5, t1;    \n\t"
		"not.b32 t6, %9;        \n\t"
		"and.b32 t6, t5, t6;    \n\t"
		"xor.b32 t7, t2, t6;    \n\t"
		"not.b32 t2, t7;        \n\t"
		"and.b32 t2, t4, t2;    \n\t"

		"and.b32 t6, %9, t4;    \n\t"
		"or.b32  t8, %7, t6;    \n\t"
		"and.b32 t6, %4, t8;    \n\t"
		"xor.b32 t8, t5, t6;    \n\t"
		"not.b32 t6, %8;        \n\t"
		"and.b32 t6, t4, t6;    \n\t"
		"xor.b32 t9, t6, t8;    \n\t"
		"xor.b32 %3, %3, t9;    \n\t"

		"and.b32 t6, t1, t0;    \n\t"
		"xor.b32 t0, %4, %7;    \n\t"
		"xor.b32 t9, t7, t0;    \n\t"
		"or.b32  t7, %6, t9;    \n\t"
		"not.b32 t9, t6;        \n\t"
		"and.b32 t9, t7, t9;    \n\t"

		"or.b32  t6, t3, t0;    \n\t"
		"not.b32 t0, t6;        \n\t"
		"and.b32 t0, t8, t0;    \n\t"
		"and.b32 t7, %7, %9;    \n\t"
		"not.b32 t8, %5;        \n\t"
		"and.b32 t8, t7, t8;    \n\t"
		"xor.b32 t10, t0, t8;   \n\t"

		"not.b32 t0, %6;        \n\t"
		"and.b32 t0, t10, t0;   \n\t"
		"or.b32  t8, t5, t7;    \n\t"
		"not.b32 t7, t0;        \n\t"
		"and.b32 t7, t8, t7;    \n\t"
		"xor.b32 t0, %4, t7;    \n\t"
		"and.b32 t7, t9, %8;    \n\t"
		"xor.b32 t8, t7, t0;    \n\t"
		"xor.b32 %1, %1, t8;    \n\t"

		"not.b32 t0, %5;        \n\t"
		"and.b32 t0, t4, t0;    \n\t"
		"not.b32 t4, %6;        \n\t"
		"and.b32 t4, t0, t4;    \n\t"
		"xor.b32 t7, t5, t6;    \n\t"
		"not.b32 t6, t7;        \n\t"
		"xor.b32 t7, t4, t6;    \n\t"
		"not.b32 t4, t2;        \n\t"
		"and.b32 t4, %8, t4;    \n\t"
		"xor.b32 t2, t4, t7;    \n\t"
		"xor.b32 %0, %0, t2;    \n\t"

		"and.b32 t2, %7, t1;    \n\t"
		"or.b32  t1, t5, t7;    \n\t"
		"not.b32 t4, t2;        \n\t"
		"and.b32 t4, t1, t4;    \n\t"
		"or.b32  t1, t3, t0;    \n\t"
		"xor.b32 t0, t4, t1;    \n\t"
		"or.b32  t1, t10, %8;   \n\t"
		"xor.b32 t2, t1, t0;    \n\t"
		"xor.b32 %2, %2, t2;    \n\t"
		
		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s4, 33 gates, 11/12 regs, 9 andn, 2/21/53/86/119 stalls, 52 biop
//
DES_SBOX_FUNCTION_QUALIFIERS void
s4(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"

	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	
		"xor.b32 t0, %4, %6;    \n\t"
		"xor.b32 t1, %6, %8;    \n\t"
		"or.b32  t2, %5, %7;    \n\t"
		"xor.b32 t3, %8, t2;    \n\t"
		"not.b32 t2, t3;        \n\t"
		"and.b32 t2, t1, t2;    \n\t"
		"not.b32 t3, %5;        \n\t"
		"and.b32 t3, t1, t3;    \n\t"
		"xor.b32 t4, %7, t3;    \n\t"
		"or.b32  t5, t0, t4;    \n\t"
		"not.b32 t6, t2;        \n\t"
		"and.b32 t6, t5, t6;    \n\t"
		"xor.b32 t5, %5, t6;    \n\t"

		"and.b32 t7, t4, t5;    \n\t"
		"not.b32 t4, t7;        \n\t"
		"and.b32 t4, t1, t4;    \n\t"
		"xor.b32 t1, t0, t5;    \n\t"
		"not.b32 t0, t4;        \n\t"
		"and.b32 t0, t1, t0;    \n\t"
		"xor.b32 t4, t2, t0;    \n\t"

		"xor.b32 t0, %5, %7;    \n\t"
		"or.b32  t2, %8, t3;    \n\t"
		"xor.b32 t3, t1, t2;    \n\t"
		"not.b32 t1, t0;        \n\t"
		"and.b32 t1, t3, t1;    \n\t"
		"xor.b32 t2, t6, t1;    \n\t"
		"not.b32 t1, t4;        \n\t"
		"and.b32 t1, %9, t1;    \n\t"
		"xor.b32 t6, t1, t2;    \n\t"
		"xor.b32 %0, %0, t6;    \n\t"

		"not.b32 t1, t2;        \n\t"
		"not.b32 t2, %9;        \n\t"
		"and.b32 t2, t4, t2;    \n\t"
		"xor.b32 t6, t2, t1;    \n\t"
		"xor.b32 %1, %1, t6;    \n\t"

		"xor.b32 t2, t4, t1;    \n\t"
		"not.b32 t1, t0;        \n\t"
		"and.b32 t1, t2, t1;    \n\t"
		"or.b32  t0, t7, t1;    \n\t"
		"xor.b32 t1, t3, t0;    \n\t"
		"or.b32  t0, t5, %9;    \n\t"
		"xor.b32 t2, t0, t1;    \n\t"
		"xor.b32 %2, %2, t2;    \n\t"

		"and.b32 t0, %9, t5;    \n\t"
		"xor.b32 t2, t0, t1;    \n\t"
		"xor.b32 %3, %3, t2;    \n\t"
		
		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s5-04832, 48 gates, 15/16 regs, 9 andn, 5/23/62/109/159 stalls, 72 biop
// Currently used for MMX/SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s5(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"

	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;          \n\t"
	    ".reg .u32 t11;          \n\t"
	    ".reg .u32 t12;          \n\t"
	
		"or.b32 t1, %4, %6; \n\t"
		"not.b32 t10, %9; \n\t"
		"and.b32 t10, t1, t10; \n\t"
		"xor.b32 t6, %4, t10; \n\t"
		"xor.b32 t2, %6, t6; \n\t"
		"or.b32 t3, %7, t2; \n\t"
		"not.b32 t7, %7; \n\t"
		"and.b32 t7, t10, t7; \n\t"
		"xor.b32 t10, %6, t7; \n\t"
		"and.b32 t7, %8, t10; \n\t"
		"or.b32 t12, %4, t2; \n\t"
		"xor.b32 t2, t7, t12; \n\t"
		"xor.b32 t7, %7, t2; \n\t"
		"xor.b32 t2, %9, t7; \n\t"
		"or.b32 t4, t6, t2; \n\t"
		"and.b32 t8, %8, t4; \n\t"
		"xor.b32 t11, t6, t8; \n\t"
		"and.b32 t9, %7, t12; \n\t"
		"xor.b32 t5, t11, t9; \n\t"
		"not.b32 t11, %4; \n\t"
		"and.b32 t11, t4, t11; \n\t"
		"xor.b32 t4, t10, t11; \n\t"
		"xor.b32 t9, %8, t3; \n\t"
		"not.b32 t0, t4; \n\t"
		"and.b32 t0, t9, t0; \n\t"
		"not.b32 t4, t0; \n\t"
		"not.b32 t0, %5; \n\t"
		"and.b32 t0, t4, t0; \n\t"
		"xor.b32 t4, t0, t7; \n\t"
		"xor.b32 %2, %2, t4; \n\t"
		"not.b32 t7, t8; \n\t"
		"and.b32 t7, t10, t7; \n\t"
		"xor.b32 t0, t11, t9; \n\t"
		"or.b32 t11, t5, t0; \n\t"
		"not.b32 t4, t7; \n\t"
		"and.b32 t4, t11, t4; \n\t"
		"not.b32 t0, t4; \n\t"
		"and.b32 t0, t3, t0; \n\t"
		"and.b32 t11, t2, t4; \n\t"
		"xor.b32 t7, t9, t11; \n\t"
		"and.b32 t2, t10, t12; \n\t"
		"or.b32 t11, t7, t2; \n\t"
		"xor.b32 t9, t8, t11; \n\t"
		"and.b32 t11, t9, %5; \n\t"
		"xor.b32 t12, t11, t5; \n\t"
		"xor.b32 %3, %3, t12; \n\t"
		"xor.b32 t12, t1, t4; \n\t"
		"xor.b32 t2, %4, t12; \n\t"
		"and.b32 t11, %7, t7; \n\t"
		"xor.b32 t8, t2, t11; \n\t"
		"or.b32 t12, t0, %5; \n\t"
		"xor.b32 t11, t12, t8; \n\t"
		"xor.b32 %0, %0, t11; \n\t"
		"xor.b32 t9, t3, t10; \n\t"
		"not.b32 t5, t8; \n\t"
		"and.b32 t5, t9, t5; \n\t"
		"xor.b32 t4, t6, t7; \n\t"
		"xor.b32 t1, t5, t4; \n\t"
		"and.b32 t2, t3, %5; \n\t"
		"xor.b32 t0, t2, t1; \n\t"
		"xor.b32 %1, %1, t0; \n\t"

		"}                      \n\t"

	    : "+r"(*out1), // %0
	      "+r"(*out2), // %1
	      "+r"(*out3), // %2
	      "+r"(*out4)  // %3
	      
	    : "r"(a1)      // %4
	      "r"(a2)      // %5
	      "r"(a3)      // %6
	      "r"(a4)      // %7
	      "r"(a5)      // %8
	      "r"(a6));    // %9
}

//
// s6-000007, 46 gates, 19 regs, 8 andn, 3/19/39/66/101 stalls, 69 biop
// Currently used for x86-64 SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s6(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    ".reg .u32 t10;         \n\t"
	    ".reg .u32 t11;         \n\t"
	    ".reg .u32 t12;         \n\t"
	    ".reg .u32 t13;         \n\t"
	    
		"xor.b32 t0, %5, %8; \n\t"

		"or.b32 t8, %5, %9; \n\t"
		"and.b32 t1, %4, t8; \n\t"
		"xor.b32 t8, t0, t1; \n\t"
		"xor.b32 t0, %9, t8; \n\t"
		"not.b32 t12, t0; \n\t"
		"and.b32 t12, %8, t12; \n\t"

		"and.b32 t11, %4, t0; \n\t"
		"xor.b32 t0, %5, t11; \n\t"
		"xor.b32 t4, %4, %6; \n\t"
		"or.b32 t13, t0, t4; \n\t"
		"xor.b32 t2, t8, t13; \n\t"

		"and.b32 t7, %6, t2; \n\t"
		"not.b32 t6, %9; \n\t"
		"and.b32 t6, t7, t6; \n\t"
		"or.b32 t9, t12, t0; \n\t"
		"xor.b32 t0, t6, t9; \n\t"
		"and.b32 t10, t0, %7; \n\t"
		"xor.b32 t5, t10, t2; \n\t"
		"xor.b32 %3, %3, t5; \n\t"

		"xor.b32 t5, %5, t13; \n\t"
		"not.b32 t13, t5; \n\t"
		"and.b32 t13, %9, t13; \n\t"
		"xor.b32 t10, %6, t13; \n\t"
		"not.b32 t13, t7; \n\t"
		"and.b32 t13, %8, t13; \n\t"
		"or.b32 t3, t10, t13; \n\t"

		"or.b32 t13, %4, t2; \n\t"
		"and.b32 t2, t9, t13; \n\t"
		"xor.b32 t9, t10, t2; \n\t"
		"not.b32 t13, t6; \n\t"
		"and.b32 t13, t9, t13; \n\t"
		"or.b32 t6, t12, %7; \n\t"
		"xor.b32 t12, t6, t13; \n\t"
		"xor.b32 %2, %2, t12; \n\t"

		"or.b32 t2, %5, t4; \n\t"
		"xor.b32 t6, t0, t2; \n\t"
		"or.b32 t12, t1, t3; \n\t"
		"xor.b32 t13, t6, t12; \n\t"

		"xor.b32 t4, t8, t9; \n\t"
		"not.b32 t0, t4; \n\t"
		"and.b32 t0, %8, t0; \n\t"
		"not.b32 t1, t5; \n\t"
		"xor.b32 t6, t2, t1; \n\t"
		"xor.b32 t12, t0, t6; \n\t"
		"not.b32 t9, %7; \n\t"
		"and.b32 t9, t12, t9; \n\t"
		"xor.b32 t12, t9, t13; \n\t"
		"xor.b32 %1, %1, t12; \n\t"

		"xor.b32 t9, %9, t11; \n\t"
		"xor.b32 t8, %4, t10; \n\t"
		"and.b32 t4, t9, t8; \n\t"
		"xor.b32 t5, t7, t6; \n\t"
		"xor.b32 t2, t4, t5; \n\t"
		"not.b32 t1, %7; \n\t"
		"and.b32 t1, t3, t1; \n\t"
		"xor.b32 t0, t1, t2; \n\t"
		"xor.b32 %0, %0, t0; \n\t"

	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(a1)     // %4
	      "r"(a2)     // %5
	      "r"(a3)     // %6
	      "r"(a4)     // %7
	      "r"(a5)     // %8
	      "r"(a6));   // %9
}

//
// s7-056945, 46 gates, 16 regs, 7 andn, 10/31/62/107/156 stalls, 67 biop
// Currently used for MMX/SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s7(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    
		"xor.b32 t6, %7, %8; \n\t"
		"xor.b32 t3, %6, t6; \n\t"
		"and.b32 t1, %9, t3; \n\t"
		"and.b32 t2, %7, t6; \n\t"
		"xor.b32 t4, %5, t2; \n\t"
		"and.b32 t0, t1, t4; \n\t"

		"and.b32 t7, %9, t2; \n\t"
		"xor.b32 t5, %6, t7; \n\t"
		"or.b32 t7, t4, t5; \n\t"
		"xor.b32 t8, %9, t6; \n\t"
		"xor.b32 t6, t7, t8; \n\t"
		"not.b32 t7, t0; \n\t"
		"and.b32 t7, %4, t7; \n\t"
		"xor.b32 t9, t7, t6; \n\t"
		"xor.b32 %3, %3, t9; \n\t"

		"not.b32 t7, t3; \n\t"
		"and.b32 t7, %8, t7; \n\t"
		"or.b32 t0, t4, t7; \n\t"
		"xor.b32 t9, t1, t5; \n\t"
		"xor.b32 t5, t0, t9; \n\t"

		"xor.b32 t0, t1, t8; \n\t"
		"not.b32 t1, t0; \n\t"
		"and.b32 t1, %7, t1; \n\t"
		"not.b32 t0, t1; \n\t"
		"and.b32 t0, t4, t0; \n\t"
		"xor.b32 t4, %8, t9; \n\t"
		"xor.b32 t1, t0, t4; \n\t"

		"or.b32 t9, t2, t6; \n\t"
		"and.b32 t0, %6, t1; \n\t"
		"or.b32 t4, t9, t0; \n\t"
		"not.b32 t2, t8; \n\t"
		"and.b32 t2, t3, t2; \n\t"
		"xor.b32 t6, t4, t2; \n\t"
		"not.b32 t8, %4; \n\t"
		"and.b32 t8, t6, t8; \n\t"
		"xor.b32 t9, t8, t5; \n\t"
		"xor.b32 %0, %0, t9; \n\t"

		"or.b32 t9, t1, t6; \n\t"
		"and.b32 t8, %9, t9; \n\t"
		"and.b32 t3, %5, t8; \n\t"
		"xor.b32 t4, t5, t6; \n\t"
		"xor.b32 t2, t3, t4; \n\t"

		"or.b32 t9, t0, t2; \n\t"
		"xor.b32 t5, t8, t9; \n\t"
		"xor.b32 t3, %8, t4; \n\t"
		"or.b32 t0, t5, t3; \n\t"
		"and.b32 t9, t0, %4; \n\t"
		"xor.b32 t5, t9, t1; \n\t"
		"xor.b32 %2, %2, t5; \n\t"

		"xor.b32 t9, t8, t0; \n\t"
		"or.b32 t4, t7, t9; \n\t"
		"not.b32 t5, t6; \n\t"
		"xor.b32 t3, t4, t5; \n\t"
		"not.b32 t1, %4; \n\t"
		"and.b32 t1, t3, t1; \n\t"
		"xor.b32 t0, t1, t2; \n\t"
		"xor.b32 %1, %1, t0; \n\t"


	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(a1)     // %4
	      "r"(a2)     // %5
	      "r"(a3)     // %6
	      "r"(a4)     // %7
	      "r"(a5)     // %8
	      "r"(a6));   // %9
}

//
// s8-004798, 41 gates, 14 regs, 7 andn, 7/35/76/118/160 stalls, 59 biop
// Currently used for MMX/SSE2
//
DES_SBOX_FUNCTION_QUALIFIERS void
s8(DES_Vector a1, DES_Vector a2, DES_Vector a3, DES_Vector a4, DES_Vector a5, DES_Vector a6,
    DES_Vector * out1, DES_Vector * out2, DES_Vector * out3, DES_Vector * out4)
{
	asm("{                      \n\t"
	    ".reg .u32 t0;          \n\t"
	    ".reg .u32 t1;          \n\t"
	    ".reg .u32 t2;          \n\t"
	    ".reg .u32 t3;          \n\t"
	    ".reg .u32 t4;          \n\t"
	    ".reg .u32 t5;          \n\t"
	    ".reg .u32 t6;          \n\t"
	    ".reg .u32 t7;          \n\t"
	    ".reg .u32 t8;          \n\t"
	    ".reg .u32 t9;          \n\t"
	    
		"not.b32 t8, %5; \n\t"
		"and.b32 t8, %6, t8; \n\t"
		"not.b32 t1, %6; \n\t"
		"and.b32 t1, %8, t1; \n\t"
		"xor.b32 t6, %7, t1; \n\t"
		"and.b32 t1, %4, t6; \n\t"
		"not.b32 t7, t8; \n\t"
		"and.b32 t7, t1, t7; \n\t"

		"not.b32 t3, t6; \n\t"
		"and.b32 t3, %5, t3; \n\t"
		"or.b32 t9, %4, t3; \n\t"
		"not.b32 t0, %6; \n\t"
		"and.b32 t0, %5, t0; \n\t"
		"xor.b32 t4, %8, t0; \n\t"
		"and.b32 t0, t9, t4; \n\t"
		"or.b32 t2, t1, t0; \n\t"

		"xor.b32 t1, t6, t0; \n\t"
		"not.b32 t0, t1; \n\t"
		"not.b32 t6, t9; \n\t"
		"and.b32 t6, %6, t6; \n\t"
		"xor.b32 t1, t0, t6; \n\t"
		"xor.b32 t9, t8, t1; \n\t"
		"or.b32 t8, t7, %9; \n\t"
		"xor.b32 t6, t8, t9; \n\t"
		"xor.b32 %1, %1, t6; \n\t"

		"xor.b32 t0, %4, t9; \n\t"
		"and.b32 t6, %8, t0; \n\t"
		"xor.b32 t8, %5, t1; \n\t"
		"xor.b32 t9, t6, t8; \n\t"
		"xor.b32 t1, t3, t9; \n\t"

		"or.b32 t6, %7, t8; \n\t"
		"xor.b32 t3, t1, t6; \n\t"
		"xor.b32 t5, t4, t3; \n\t"
		"xor.b32 t8, %4, t5; \n\t"
		"and.b32 t6, t8, %9; \n\t"
		"xor.b32 t4, t6, t1; \n\t"
		"xor.b32 %3, %3, t4; \n\t"

		"xor.b32 t6, t2, t9; \n\t"
		"or.b32 t4, %5, t6; \n\t"
		"xor.b32 t3, t0, t4; \n\t"
		"xor.b32 t8, %8, t3; \n\t"
		"and.b32 t9, t2, %9; \n\t"
		"xor.b32 t6, t9, t8; \n\t"
		"xor.b32 %2, %2, t6; \n\t"

		"or.b32  t9, %7, t0; \n\t"
		"not.b32 t6, t9; \n\t"
		"and.b32 t6, t8, t6; \n\t"
		"or.b32  t4, t7, t6; \n\t"
		"xor.b32 t3, t5, t4; \n\t"
		"or.b32  t2, t3, %9; \n\t"
		"xor.b32 t0, t2, t1; \n\t"
		"xor.b32 %0, %0, t0; \n\t"

	    "}                      \n\t"

	    : "+r"(*out1),  // %0
	      "+r"(*out2),  // %1
	      "+r"(*out3),  // %2
	      "+r"(*out4)   // %3
	      
	    : "r"(a1)     // %4
	      "r"(a2)     // %5
	      "r"(a3)     // %6
	      "r"(a4)     // %7
	      "r"(a5)     // %8
	      "r"(a6));   // %9
}

#define CLEAR_BLOCK_8(i)                                                             \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 0, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 1, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 2, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 3, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 4, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 5, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 6, DES_VECTOR_ZERO); \
	DES_VECTOR_SET(DESContextArray[threadIdx.x].dataBlocks[i] , 7, DES_VECTOR_ZERO); \

#define CLEAR_BLOCK()  \
	CLEAR_BLOCK_8(0);  \
	CLEAR_BLOCK_8(8);  \
	CLEAR_BLOCK_8(16); \
	CLEAR_BLOCK_8(24); \
	CLEAR_BLOCK_8(32); \
	CLEAR_BLOCK_8(40); \
	CLEAR_BLOCK_8(48); \
	CLEAR_BLOCK_8(56); \

#define x(p)    DES_VECTOR_XOR_FUNC(DESContextArray[threadIdx.x].dataBlocks[CUDA_expansionFunction[p]], DESContextArray[threadIdx.x].keys[keySchedule[keyScheduleIndexBase + (p)]])
#define y(p, q) DES_VECTOR_XOR_FUNC(DESContextArray[threadIdx.x].dataBlocks[p],                         DESContextArray[threadIdx.x].keys[keySchedule[keyScheduleIndexBase + (q)]])
#define z(r)    (&DESContextArray[threadIdx.x].dataBlocks[r])

DES_FUNCTION_QUALIFIERS
void DES_Crypt()
{
	unsigned char iterations;
	short roundsAndSwapped; 
	short keyScheduleIndexBase = 0;

	if (threadIdx.y == 0)
		CLEAR_BLOCK();
	
	roundsAndSwapped = 8;
	iterations = 25;

start:
	__syncthreads();
	switch (threadIdx.y) {
	case 0: s1(x( 0),     x( 1),     x( 2),     x( 3),     x( 4),     x( 5),     z(40), z(48), z(54), z(62)); break;
	case 1: s2(x( 6),     x( 7),     x( 8),     x( 9),     x(10),     x(11),     z(44), z(59), z(33), z(49)); break;
	case 2: s3(y( 7, 12), y( 8, 13), y( 9, 14), y(10, 15), y(11, 16), y(12, 17), z(55), z(47), z(61), z(37)); break;
	case 3: s4(y(11, 18), y(12, 19), y(13, 20), y(14, 21), y(15, 22), y(16, 23), z(57), z(51), z(41), z(32)); break;
	case 4: s5(x(24),     x(25),     x(26),     x(27),     x(28),     x(29),     z(39), z(45), z(56), z(34)); break;
	case 5: s6(x(30),     x(31),     x(32),     x(33),     x(34),     x(35),     z(35), z(60), z(42), z(50)); break;
	case 6: s7(y(23, 36), y(24, 37), y(25, 38),	y(26, 39), y(27, 40), y(28, 41), z(63), z(43), z(53), z(38)); break;
	case 7: s8(y(27, 42), y(28, 43), y(29, 44), y(30, 45), y(31, 46), y( 0, 47), z(36), z(58), z(46), z(52)); break;
	}
	__syncthreads();

	if (roundsAndSwapped == 0x100)
		goto next;

swap:
	__syncthreads();
	switch (threadIdx.y) {
	case 0: s1(x(48),     x(49),     x(50),     x(51),     x(52),     x(53),     z( 8), z(16), z(22), z(30)); break;
	case 1: s2(x(54),     x(55),     x(56),     x(57),     x(58),     x(59),     z(12), z(27), z( 1), z(17)); break;
	case 2: s3(y(39, 60), y(40, 61), y(41, 62), y(42, 63), y(43, 64), y(44, 65), z(23), z(15), z(29), z( 5)); break;
	case 3: s4(y(43, 66), y(44, 67), y(45, 68), y(46, 69), y(47, 70), y(48, 71), z(25), z(19), z( 9), z( 0)); break;
	case 4: s5(x(72),     x(73),     x(74),     x(75),     x(76),     x(77),     z( 7), z(13), z(24), z( 2)); break;
	case 5: s6(x(78),     x(79),     x(80),     x(81),     x(82),     x(83),     z( 3), z(28), z(10), z(18)); break;
	case 6: s7(y(55, 84), y(56, 85), y(57, 86), y(58, 87), y(59, 88), y(60, 89), z(31), z(11), z(21), z( 6)); break;
	case 7: s8(y(59, 90), y(60, 91), y(61, 92), y(62, 93), y(63, 94), y(32, 95), z( 4), z(26), z(14), z(20)); break;
	}	
	__syncthreads();

	keyScheduleIndexBase += 96;

	if (--roundsAndSwapped)
		goto start;
	keyScheduleIndexBase -= (0x300 + 48);
	roundsAndSwapped = 0x108;
	if (--iterations)
		goto swap;
	return;

next:
	keyScheduleIndexBase -= (0x300 - 48);
	roundsAndSwapped = 8;
	iterations--;
	goto start;
}

#define GET_TRIPCODE_CHAR_INDEX(r, t, i0, i1, i2, i3, i4, i5, pos)  \
		(  ((((r)[i0] & (0x01 << (t))) ? (0x1) : (0x0)) << (5 + ((pos) * 6)))  \
	 	 | ((((r)[i1] & (0x01 << (t))) ? (0x1) : (0x0)) << (4 + ((pos) * 6)))  \
		 | ((((r)[i2] & (0x01 << (t))) ? (0x1) : (0x0)) << (3 + ((pos) * 6)))  \
		 | ((((r)[i3] & (0x01 << (t))) ? (0x1) : (0x0)) << (2 + ((pos) * 6)))  \
		 | ((((r)[i4] & (0x01 << (t))) ? (0x1) : (0x0)) << (1 + ((pos) * 6)))  \
		 | ((((r)[i5] & (0x01 << (t))) ? (0x1) : (0x0)) << (0 + ((pos) * 6)))) \

#define GET_TRIPCODE_CHAR_INDEX_LAST(r, t, i0, i1, i2, i3)     \
		(  ((((r)[i0] & (0x01 << (t))) ? (0x1) : (0x0)) << 5)  \
	 	 | ((((r)[i1] & (0x01 << (t))) ? (0x1) : (0x0)) << 4)  \
		 | ((((r)[i2] & (0x01 << (t))) ? (0x1) : (0x0)) << 3)  \
		 | ((((r)[i3] & (0x01 << (t))) ? (0x1) : (0x0)) << 2)) \

DES_FUNCTION_QUALIFIERS void
DES_GetTripcodeChunks(int tripcodeIndex, unsigned int *tripcodeChunkArray, int searchMode)
{
	// Perform the final permutation here.
	if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
	} else if (searchMode == SEARCH_MODE_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
		                        | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
		                        | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
		                        | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
		                        | GET_TRIPCODE_CHAR_INDEX_LAST(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] =   GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 4)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 3)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 2)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 1)
								| GET_TRIPCODE_CHAR_INDEX_LAST(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	} else {
		tripcodeChunkArray[0] =   GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 4)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 3)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 2)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 1)
								| GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0);
		tripcodeChunkArray[1] = ((tripcodeChunkArray[0] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0);
		tripcodeChunkArray[2] = ((tripcodeChunkArray[1] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0);
		tripcodeChunkArray[3] = ((tripcodeChunkArray[2] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0);
		tripcodeChunkArray[4] = ((tripcodeChunkArray[3] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0);
		tripcodeChunkArray[5] = ((tripcodeChunkArray[4] << 6) & 0x3fffffff) | GET_TRIPCODE_CHAR_INDEX_LAST(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 48, 16, 56, 24);
	}
}

DES_FUNCTION_QUALIFIERS
unsigned char *DES_GetTripcode(int tripcodeIndex, unsigned char *tripcode)
{
	// Perform the final permutation as necessary.
  	tripcode[0] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0)];
  	tripcode[1] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0)];
  	tripcode[2] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0)];
  	tripcode[3] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0)];
  	tripcode[4] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0)];
  	tripcode[5] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0)];
  	tripcode[6] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0)];
  	tripcode[7] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0)];
  	tripcode[8] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0)];
	tripcode[9] = CUDA_DES_indexToCharTable[GET_TRIPCODE_CHAR_INDEX_LAST(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 48, 16, 56, 24)];
 	tripcode[10] = '\0';

	return tripcode;
}

#define SET_KEY_CHAR(var, flag, table, value)             \
	if (!(flag)) {                                        \
		var = (table)[(value)];                           \
		isSecondByte = IS_FIRST_BYTE_SJIS(var);           \
	} else {                                              \
		var = CUDA_keyCharTable_SecondByte[(value)];          \
		isSecondByte = FALSE;                             \
	}

#define CUDA_DES_DEFINE_SEARCH_FUNCTION(functionName) \
__global__ void functionName(\
	GPUOutput *outputArray,\
	unsigned char      *keyBitmap,\
	unsigned int     *tripcodeChunkArray,\
	unsigned int      numTripcodeChunk,\
	int         searchMode) {

#define SET_ALL_BITS_FOR_KEY(i, j, k)                                  \
	if (key[j] & (0x1 << (k)))                                         \
		DESContextArray[threadIdx.x].keys[i] = 0xffffffff;             \

#define SET_BIT_FOR_KEY(i, j, k)                                       \
	if (key[j] & (0x1 << (k)))                                         \
		DESContextArray[threadIdx.x].keys[i] |= 0x1 << tripcodeIndex;  \

#define CUDA_DES_BEFORE_SEARCHING \
	GPUOutput  *output = &outputArray[blockIdx.x * CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK + threadIdx.x];\
	unsigned char        key[8];\
	BOOL         isSecondByte, isKey6SecondByte;\
	unsigned char        tripcodeIndex;\
	unsigned char        passCount;\
	\
	if (threadIdx.y == 0) {\
		output->numMatchingTripcodes = 0;\
		key[0] = CUDA_key[0];\
		key[1] = CUDA_key[1];\
		key[2] = CUDA_key[2];\
		isSecondByte = IS_FIRST_BYTE_SJIS(CUDA_key[2]);\
		SET_KEY_CHAR(key[3], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[3] + ((blockIdx.x  >> 6) & 63));\
		SET_KEY_CHAR(key[4], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[4] + ( blockIdx.x        & 63));\
		SET_KEY_CHAR(key[5], isSecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[5] + ( threadIdx.x       & 63));\
		isKey6SecondByte = isSecondByte;\
	}\
		                            \
	DES_ClearKeys(0);               \
	SET_ALL_BITS_FOR_KEY( 0, 0, 0); \
	SET_ALL_BITS_FOR_KEY( 1, 0, 1); \
	SET_ALL_BITS_FOR_KEY( 2, 0, 2); \
	SET_ALL_BITS_FOR_KEY( 3, 0, 3); \
	SET_ALL_BITS_FOR_KEY( 4, 0, 4); \
	SET_ALL_BITS_FOR_KEY( 5, 0, 5); \
	SET_ALL_BITS_FOR_KEY( 6, 0, 6); \
		                            \
	DES_ClearKeys(1);               \
	SET_ALL_BITS_FOR_KEY( 7, 1, 0); \
	SET_ALL_BITS_FOR_KEY( 8, 1, 1); \
	SET_ALL_BITS_FOR_KEY( 9, 1, 2); \
	SET_ALL_BITS_FOR_KEY(10, 1, 3); \
	SET_ALL_BITS_FOR_KEY(11, 1, 4); \
	SET_ALL_BITS_FOR_KEY(12, 1, 5); \
	SET_ALL_BITS_FOR_KEY(13, 1, 6); \
		                            \
	DES_ClearKeys(2);               \
	SET_ALL_BITS_FOR_KEY(14, 2, 0); \
	SET_ALL_BITS_FOR_KEY(15, 2, 1); \
	SET_ALL_BITS_FOR_KEY(16, 2, 2); \
	SET_ALL_BITS_FOR_KEY(17, 2, 3); \
	SET_ALL_BITS_FOR_KEY(18, 2, 4); \
	SET_ALL_BITS_FOR_KEY(19, 2, 5); \
	SET_ALL_BITS_FOR_KEY(20, 2, 6); \
		                            \
	DES_ClearKeys(3);               \
	SET_ALL_BITS_FOR_KEY(21, 3, 0); \
	SET_ALL_BITS_FOR_KEY(22, 3, 1); \
	SET_ALL_BITS_FOR_KEY(23, 3, 2); \
	SET_ALL_BITS_FOR_KEY(24, 3, 3); \
	SET_ALL_BITS_FOR_KEY(25, 3, 4); \
	SET_ALL_BITS_FOR_KEY(26, 3, 5); \
	SET_ALL_BITS_FOR_KEY(27, 3, 6); \
		                            \
	DES_ClearKeys(4);               \
	SET_ALL_BITS_FOR_KEY(28, 4, 0); \
	SET_ALL_BITS_FOR_KEY(29, 4, 1); \
	SET_ALL_BITS_FOR_KEY(30, 4, 2); \
	SET_ALL_BITS_FOR_KEY(31, 4, 3); \
	SET_ALL_BITS_FOR_KEY(32, 4, 4); \
	SET_ALL_BITS_FOR_KEY(33, 4, 5); \
	SET_ALL_BITS_FOR_KEY(34, 4, 6); \
		                            \
	DES_ClearKeys(5);               \
	SET_ALL_BITS_FOR_KEY(35, 5, 0); \
	SET_ALL_BITS_FOR_KEY(36, 5, 1); \
	SET_ALL_BITS_FOR_KEY(37, 5, 2); \
	SET_ALL_BITS_FOR_KEY(38, 5, 3); \
	SET_ALL_BITS_FOR_KEY(39, 5, 4); \
	SET_ALL_BITS_FOR_KEY(40, 5, 5); \
	SET_ALL_BITS_FOR_KEY(41, 5, 6); \
		                            \
	for (passCount = 0; passCount < CUDA_DES_MAX_PASS_COUNT; ++passCount) {\
		__syncthreads();\
		\
		if (threadIdx.y == 0) {\
			SET_KEY_CHAR(key[6], isKey6SecondByte, CUDA_keyCharTable_FirstByte, CUDA_key[6] + ((passCount & 0xf)  | ((threadIdx.x & 0xc0) >> 2)));\
			DES_ClearKeys(6);               \
			SET_ALL_BITS_FOR_KEY(42, 6, 0);      \
			SET_ALL_BITS_FOR_KEY(43, 6, 1);      \
			SET_ALL_BITS_FOR_KEY(44, 6, 2);      \
			SET_ALL_BITS_FOR_KEY(45, 6, 3);      \
			SET_ALL_BITS_FOR_KEY(46, 6, 4);      \
			SET_ALL_BITS_FOR_KEY(47, 6, 5);      \
			SET_ALL_BITS_FOR_KEY(48, 6, 6);      \
			\
			DES_ClearKeys(7);               \
			if (isSecondByte) {\
				for (tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {\
					key[7] = CUDA_keyCharTable_SecondByte[CUDA_key[7] + (tripcodeIndex & 63)];\
					SET_BIT_FOR_KEY(49, 7, 0);      \
					SET_BIT_FOR_KEY(50, 7, 1);      \
					SET_BIT_FOR_KEY(51, 7, 2);      \
					SET_BIT_FOR_KEY(52, 7, 3);      \
					SET_BIT_FOR_KEY(53, 7, 4);      \
					SET_BIT_FOR_KEY(54, 7, 5);      \
					SET_BIT_FOR_KEY(55, 7, 6);      \
				}\
			} else {\
				for (tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {\
					key[7] = CUDA_keyCharTable_FirstByte [CUDA_key[7] + (tripcodeIndex & 63)];\
					SET_BIT_FOR_KEY(49, 7, 0);      \
					SET_BIT_FOR_KEY(50, 7, 1);      \
					SET_BIT_FOR_KEY(51, 7, 2);      \
					SET_BIT_FOR_KEY(52, 7, 3);      \
					SET_BIT_FOR_KEY(53, 7, 4);      \
					SET_BIT_FOR_KEY(54, 7, 5);      \
					SET_BIT_FOR_KEY(55, 7, 6);      \
				}\
			}\
		}\
		\
		__syncthreads();\
		DES_Crypt();\
		\
		__syncthreads();\
		if (threadIdx.y == 0) {\
			BOOL found = FALSE;\
			for (tripcodeIndex = 0; tripcodeIndex < CUDA_DES_BS_DEPTH; ++tripcodeIndex) {

#define CUDA_DES_END_OF_SEAERCH_FUNCTION \
			}\
quit_loops:\
			if (found == TRUE) {\
				output->numMatchingTripcodes  = 1;\
				DES_GetTripcode(tripcodeIndex, output->pair.tripcode.c);\
				output->pair.key.c[0] = key[0];\
				output->pair.key.c[1] = key[1];\
				output->pair.key.c[2] = key[2];\
				output->pair.key.c[3] = key[3];\
				output->pair.key.c[4] = key[4];\
				output->pair.key.c[5] = key[5];\
				output->pair.key.c[6] = key[6];\
				output->pair.key.c[7] = (isSecondByte)\
												? (CUDA_keyCharTable_SecondByte[CUDA_key[7] + (tripcodeIndex & 63)])\
												: (CUDA_keyCharTable_FirstByte [CUDA_key[7] + (tripcodeIndex & 63)]);\
			}\
		}\
	}\
	if (threadIdx.y == 0)\
		output->numGeneratedTripcodes = CUDA_DES_BS_DEPTH * CUDA_DES_MAX_PASS_COUNT;\
}

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == tripcodeChunk) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardOrBackwardMatching)
	unsigned int tripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, &tripcodeChunk, searchMode);
	if (keyBitmap[tripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)])
		continue;
	for (int j = 0; j < numTripcodeChunk; ++j){
		int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
		while (tripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
			middle = (lower + upper) >> 1;
			if (tripcodeChunk > tripcodeChunkArray[middle]) {
				lower = middle + 1;
			} else {
				upper = middle - 1;
			}
		}
		if (tripcodeChunk == tripcodeChunkArray[middle]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 63, 31, 38,  6, 46, 14, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 54, 22, 62, 30, 37,  5, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 45, 13, 53, 21, 61, 29, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 36,  4, 44, 12, 52, 20, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 60, 28, 35,  3, 43, 11, 0) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_BackwardMatching_1Chunk)
	unsigned int tripcodeChunk0 = tripcodeChunkArray[0];
CUDA_DES_BEFORE_SEARCHING
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 51, 19, 59, 27, 34,  2, 0) != ((tripcodeChunk0 >> (6 * 4)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 42, 10, 50, 18, 58, 26, 0) != ((tripcodeChunk0 >> (6 * 3)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 33,  1, 41,  9, 49, 17, 0) != ((tripcodeChunk0 >> (6 * 2)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 57, 25, 32,  0, 40,  8, 0) != ((tripcodeChunk0 >> (6 * 1)) & 0x3f))
		goto skip_final_permutation;
	if (GET_TRIPCODE_CHAR_INDEX_LAST(DESContextArray[threadIdx.x].dataBlocks, tripcodeIndex, 48, 16, 56, 24) != ((tripcodeChunk0 >> (6 * 0)) & 0x3f))
		goto skip_final_permutation;
	found = TRUE;
	goto quit_loops;
skip_final_permutation:
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[pos]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_Flexible)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	for (int pos = 0; pos < 6; ++pos) {
		unsigned int generatedTripcodeChunk = generatedTripcodeChunkArray[pos];
		if (keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)])
			continue;
		for (int j = 0; j < numTripcodeChunk; ++j){
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple)
	unsigned int generatedTripcodeChunkArray[6];
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[0]) {
			found = TRUE;
			goto quit_loops;
		}
	}
	//
	for (int j = 0; j < numTripcodeChunk; ++j){
		if (tripcodeChunkArray[j] == generatedTripcodeChunkArray[1]) {
			found = TRUE;
			goto quit_loops;
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION

CUDA_DES_DEFINE_SEARCH_FUNCTION(CUDA_PerformSearching_DES_ForwardAndBackwardMatching)
	unsigned int generatedTripcodeChunkArray[6];
	unsigned int generatedTripcodeChunk;
CUDA_DES_BEFORE_SEARCHING
	DES_GetTripcodeChunks(tripcodeIndex, generatedTripcodeChunkArray, searchMode);
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[0];
	if (!keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j){
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
	//
	generatedTripcodeChunk = generatedTripcodeChunkArray[1];
	if (!keyBitmap[generatedTripcodeChunk >> ((5 - KEY_BITMAP_LEN_STRING) * 6)]) {
		for (int j = 0; j < numTripcodeChunk; ++j) {
			int lower = 0, upper = numTripcodeChunk - 1, middle = lower;
			while (generatedTripcodeChunk != tripcodeChunkArray[middle] && lower <= upper) {
				middle = (lower + upper) >> 1;
				if (generatedTripcodeChunk > tripcodeChunkArray[middle]) {
					lower = middle + 1;
				} else {
					upper = middle - 1;
				}
			}
			if (generatedTripcodeChunk == tripcodeChunkArray[middle]) {
				found = TRUE;
				goto quit_loops;
			}
		}
	}
CUDA_DES_END_OF_SEAERCH_FUNCTION



///////////////////////////////////////////////////////////////////////////////
// CUDA SEARCH THREAD FOR 10 CHARACTER TRIPCODES                             //
///////////////////////////////////////////////////////////////////////////////

unsigned WINAPI Thread_SearchForDESTripcodesOnCUDADevice(LPVOID info)
{
	hipError_t     hipError_t;
	hipDeviceProp_t  CUDADeviceProperties;
	unsigned int    numBlocksPerSM;
	unsigned int    numBlocksPerGrid;
	GPUOutput      *outputArray = NULL;
	GPUOutput      *CUDA_outputArray = NULL;
	unsigned int   *CUDA_tripcodeChunkArray = NULL;
	unsigned char  *CUDA_keyBitmap = NULL;
	unsigned int    sizeOutputArray;
	unsigned char   key[MAX_LEN_TRIPCODE + 1];
	unsigned char   salt[3];
	unsigned char   expansionFunction[96];
	char            status[LEN_LINE_BUFFER_FOR_SCREEN] = "";
	int             optimizationPhase    = CUDA_OPTIMIZATION_PHASE_NUM_BLOCKS;
	int             optimizationSubphase = 0;
	double          timeElapsedInOptimizationSubphase = 0;
	static int      numBlocksTableForOptimization[] = {8, 16, 32, 48, 64, 96, 128, 160, 192, 224, 256, 0};
	double          numGeneratedTripcodes = 0;
	double          speedInPreviousSubphase = 0;
	double          speedInCurrentSubphase = 0;
	DWORD           startingTime;
	DWORD           endingTime;
	double          deltaTime;

	key[lenTripcode] = '\0';
	salt[2]          = '\0';
	
	CUDA_ERROR(hipSetDevice(((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	CUDA_ERROR(hipGetDeviceProperties(&CUDADeviceProperties, ((CUDADeviceSearchThreadInfo *)info)->CUDADeviceIndex));
	if (CUDADeviceProperties.computeMode == hipComputeModeProhibited) {
		sprintf(status, "[disabled]");
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), FALSE, status);
		return 0;
	}

	if (options.CUDANumBlocksPerSM == CUDA_NUM_BLOCKS_PER_SM_NIL) {
		numBlocksPerSM = numBlocksTableForOptimization[optimizationSubphase];
	} else {
		numBlocksPerSM = options.CUDANumBlocksPerSM;
	}
	numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
	sizeOutputArray = CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK * numBlocksPerGrid;
	outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(outputArray == NULL, ERROR_NO_MEMORY, "Not enough memory.");
	hipError_t = hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray);
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);
	hipError_t = hipMalloc((void **)&CUDA_keyBitmap, KEY_BITMAP_SIZE);
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);
	hipError_t = hipMalloc((void **)&CUDA_tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk); 
	ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
	CUDA_ERROR(hipError_t);

	CUDA_ERROR(hipMemcpy(CUDA_tripcodeChunkArray, tripcodeChunkArray, sizeof(unsigned int) * numTripcodeChunk, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpy(CUDA_keyBitmap, keyBitmap, KEY_BITMAP_SIZE, hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_base64CharTable),      base64CharTable,      sizeof(base64CharTable)));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_OneByte), keyCharTable_OneByte, SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_keyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	
	startingTime = timeGetTime();

	while (!GetTerminationState()) {
		// Choose the first 3 characters of the key.
		SetCharactersInTripcodeKey(key, 3);
		
		// Make sure that the first 3 bytes consist of valid Shift-JIS characters.
		for (int i = 3; i < lenTripcode; ++i)
			key[i] = 'A';
		if (!IsValidKey(key))
			continue;
		
		// Generate random bytes for the key to ensure the randomness of them.
		unsigned char randomByteForKey6 = RandomByte();
		for (int i = 3; i < lenTripcode; ++i)
			key[i] = RandomByte();
		
		// Create an expansion function based on the salt.
		salt[0] = CONVERT_CHAR_FOR_SALT(key[1]);
		salt[1] = CONVERT_CHAR_FOR_SALT(key[2]);
		DES_CreateExpansionFunction((char *)salt, expansionFunction);

		// Call an appropriate CUDA kernel.
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_key),               key,               lenTripcode));
		CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_expansionFunction), expansionFunction, sizeof(expansionFunction)));
		dim3 dimBlock(CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK, CUDA_DES_NUM_THREADS_FOR_BITSLICE);
		dim3 dimGrid(numBlocksPerGrid);
		if (searchMode == SEARCH_MODE_FLEXIBLE) {
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_Flexible_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					searchMode);
			} else {
				CUDA_PerformSearching_DES_Flexible<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					searchMode);
			}
		} else if (searchMode == SEARCH_MODE_FORWARD_AND_BACKWARD_MATCHING) {
			if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					searchMode);
			} else {
				CUDA_PerformSearching_DES_ForwardAndBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					searchMode);
			}
		} else {
			if (numTripcodeChunk == 1) {
				if (searchMode == SEARCH_MODE_FORWARD_MATCHING) {
					CUDA_PerformSearching_DES_ForwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
						CUDA_outputArray,
						CUDA_keyBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						searchMode);
				} else {
					CUDA_PerformSearching_DES_BackwardMatching_1Chunk<<<dimGrid, dimBlock>>>(
						CUDA_outputArray,
						CUDA_keyBitmap,
						CUDA_tripcodeChunkArray,
						numTripcodeChunk,
						searchMode);
				}
			} else if (numTripcodeChunk <= CUDA_SIMPLE_SEARCH_THRESHOLD) {
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching_Simple<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					searchMode);
			} else {
				CUDA_PerformSearching_DES_ForwardOrBackwardMatching<<<dimGrid, dimBlock>>>(
					CUDA_outputArray,
					CUDA_keyBitmap,
					CUDA_tripcodeChunkArray,
					numTripcodeChunk,
					searchMode);
			}
		}
		CUDA_ERROR(hipGetLastError());
		// CUDA_ERROR(hipDeviceSynchronize()); // Check errors at kernel launch.

		// Process the output array.
		CUDA_ERROR(hipMemcpy(outputArray, CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray, hipMemcpyDeviceToHost));
		numGeneratedTripcodes += ProcessGPUOutput(key, outputArray, sizeOutputArray, FALSE);
		
		// Optimization
		endingTime = timeGetTime();
		deltaTime = (endingTime >= startingTime)
						? ((double)endingTime - (double)startingTime                     ) * 0.001
						: ((double)endingTime - (double)startingTime + (double)0xffffffff) * 0.001;
		while (GetPauseState() && !GetTerminationState())
			Sleep(PAUSE_INTERVAL);
		startingTime = timeGetTime();
		timeElapsedInOptimizationSubphase += deltaTime;
		speedInCurrentSubphase = numGeneratedTripcodes / timeElapsedInOptimizationSubphase;
		//
		if (optimizationPhase == CUDA_OPTIMIZATION_PHASE_NUM_BLOCKS) {
			if (options.CUDANumBlocksPerSM != CUDA_NUM_BLOCKS_PER_SM_NIL) {
				optimizationPhase     = CUDA_OPTIMIZATION_PHASE_COMPLETED;
				optimizationSubphase  = 0;
				numGeneratedTripcodes = 0;
				timeElapsedInOptimizationSubphase = 0;
			} else if (timeElapsedInOptimizationSubphase >= CUDA_OPTIMIZATION_SUBPHASE_DURATION) {
				if (   optimizationSubphase > 0
				    && (   speedInPreviousSubphase > speedInCurrentSubphase
					    || fabs(speedInPreviousSubphase - speedInCurrentSubphase) / speedInPreviousSubphase < CUDA_OPTIMIZATION_THRESHOLD)) {
					numBlocksPerSM = numBlocksTableForOptimization[(speedInPreviousSubphase > speedInCurrentSubphase) ? (optimizationSubphase - 1) : (optimizationSubphase)];
					optimizationPhase = CUDA_OPTIMIZATION_PHASE_COMPLETED;
					optimizationSubphase = 0;
					numGeneratedTripcodes = 0;
				} else if (numBlocksTableForOptimization[optimizationSubphase + 1] > 0) {
					numBlocksPerSM = numBlocksTableForOptimization[++optimizationSubphase];
					timeElapsedInOptimizationSubphase = 0;
					numGeneratedTripcodes = 0;
					speedInPreviousSubphase = speedInCurrentSubphase;
				} else {
					optimizationPhase = CUDA_OPTIMIZATION_PHASE_COMPLETED;
					optimizationSubphase = 0;
					numGeneratedTripcodes = 0;
				}
				timeElapsedInOptimizationSubphase = 0;
				numGeneratedTripcodes = 0;
				numBlocksPerGrid = numBlocksPerSM * CUDADeviceProperties.multiProcessorCount;
				sizeOutputArray = CUDA_DES_NUM_BITSLICE_DES_CONTEXTS_PER_BLOCK * numBlocksPerGrid;
				free(outputArray);
				outputArray = (GPUOutput *)malloc(sizeof(GPUOutput) * sizeOutputArray);
				ERROR0(outputArray == NULL, ERROR_NO_MEMORY, "Not enough memory.");
				CUDA_ERROR(hipFree(CUDA_outputArray));
				hipError_t = hipMalloc((void **)&CUDA_outputArray, sizeof(GPUOutput) * sizeOutputArray);
				ERROR0(hipError_t == hipErrorOutOfMemory, ERROR_NO_MEMORY, "Not enough memory.");
				CUDA_ERROR(hipError_t);
			}
		}
		//
		sprintf(status,
			    "%s%.1lfM TPS, %d blocks/SM",
				(optimizationPhase != CUDA_OPTIMIZATION_PHASE_COMPLETED) ? "[optimizing...] " : "",
				speedInCurrentSubphase / 1000000,
				numBlocksPerSM);
		UpdateCUDADeviceStatus(((CUDADeviceSearchThreadInfo *)info), (optimizationPhase != CUDA_OPTIMIZATION_PHASE_COMPLETED), status);
	}

	RELEASE_AND_SET_TO_NULL(CUDA_outputArray,        hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_tripcodeChunkArray, hipFree);
	RELEASE_AND_SET_TO_NULL(CUDA_keyBitmap,          hipFree);
	RELEASE_AND_SET_TO_NULL(outputArray,             free);
}
