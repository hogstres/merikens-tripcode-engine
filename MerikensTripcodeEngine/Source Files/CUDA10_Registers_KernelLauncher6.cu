// Meriken's Tripcode Engine 2.0.0
// Copyright (c) 2011-2015 Meriken.Z. <meriken.2ch@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "CUDA10_Registers_Kernel_Common.h"

#define SALT 3072
#include "CUDA10_Registers_Kernel.h"
#define SALT 3073
#include "CUDA10_Registers_Kernel.h"
#define SALT 3074
#include "CUDA10_Registers_Kernel.h"
#define SALT 3075
#include "CUDA10_Registers_Kernel.h"
#define SALT 3076
#include "CUDA10_Registers_Kernel.h"
#define SALT 3077
#include "CUDA10_Registers_Kernel.h"
#define SALT 3078
#include "CUDA10_Registers_Kernel.h"
#define SALT 3079
#include "CUDA10_Registers_Kernel.h"
#define SALT 3080
#include "CUDA10_Registers_Kernel.h"
#define SALT 3081
#include "CUDA10_Registers_Kernel.h"
#define SALT 3082
#include "CUDA10_Registers_Kernel.h"
#define SALT 3083
#include "CUDA10_Registers_Kernel.h"
#define SALT 3084
#include "CUDA10_Registers_Kernel.h"
#define SALT 3085
#include "CUDA10_Registers_Kernel.h"
#define SALT 3086
#include "CUDA10_Registers_Kernel.h"
#define SALT 3087
#include "CUDA10_Registers_Kernel.h"
#define SALT 3088
#include "CUDA10_Registers_Kernel.h"
#define SALT 3089
#include "CUDA10_Registers_Kernel.h"
#define SALT 3090
#include "CUDA10_Registers_Kernel.h"
#define SALT 3091
#include "CUDA10_Registers_Kernel.h"
#define SALT 3092
#include "CUDA10_Registers_Kernel.h"
#define SALT 3093
#include "CUDA10_Registers_Kernel.h"
#define SALT 3094
#include "CUDA10_Registers_Kernel.h"
#define SALT 3095
#include "CUDA10_Registers_Kernel.h"
#define SALT 3096
#include "CUDA10_Registers_Kernel.h"
#define SALT 3097
#include "CUDA10_Registers_Kernel.h"
#define SALT 3098
#include "CUDA10_Registers_Kernel.h"
#define SALT 3099
#include "CUDA10_Registers_Kernel.h"
#define SALT 3100
#include "CUDA10_Registers_Kernel.h"
#define SALT 3101
#include "CUDA10_Registers_Kernel.h"
#define SALT 3102
#include "CUDA10_Registers_Kernel.h"
#define SALT 3103
#include "CUDA10_Registers_Kernel.h"
#define SALT 3104
#include "CUDA10_Registers_Kernel.h"
#define SALT 3105
#include "CUDA10_Registers_Kernel.h"
#define SALT 3106
#include "CUDA10_Registers_Kernel.h"
#define SALT 3107
#include "CUDA10_Registers_Kernel.h"
#define SALT 3108
#include "CUDA10_Registers_Kernel.h"
#define SALT 3109
#include "CUDA10_Registers_Kernel.h"
#define SALT 3110
#include "CUDA10_Registers_Kernel.h"
#define SALT 3111
#include "CUDA10_Registers_Kernel.h"
#define SALT 3112
#include "CUDA10_Registers_Kernel.h"
#define SALT 3113
#include "CUDA10_Registers_Kernel.h"
#define SALT 3114
#include "CUDA10_Registers_Kernel.h"
#define SALT 3115
#include "CUDA10_Registers_Kernel.h"
#define SALT 3116
#include "CUDA10_Registers_Kernel.h"
#define SALT 3117
#include "CUDA10_Registers_Kernel.h"
#define SALT 3118
#include "CUDA10_Registers_Kernel.h"
#define SALT 3119
#include "CUDA10_Registers_Kernel.h"
#define SALT 3120
#include "CUDA10_Registers_Kernel.h"
#define SALT 3121
#include "CUDA10_Registers_Kernel.h"
#define SALT 3122
#include "CUDA10_Registers_Kernel.h"
#define SALT 3123
#include "CUDA10_Registers_Kernel.h"
#define SALT 3124
#include "CUDA10_Registers_Kernel.h"
#define SALT 3125
#include "CUDA10_Registers_Kernel.h"
#define SALT 3126
#include "CUDA10_Registers_Kernel.h"
#define SALT 3127
#include "CUDA10_Registers_Kernel.h"
#define SALT 3128
#include "CUDA10_Registers_Kernel.h"
#define SALT 3129
#include "CUDA10_Registers_Kernel.h"
#define SALT 3130
#include "CUDA10_Registers_Kernel.h"
#define SALT 3131
#include "CUDA10_Registers_Kernel.h"
#define SALT 3132
#include "CUDA10_Registers_Kernel.h"
#define SALT 3133
#include "CUDA10_Registers_Kernel.h"
#define SALT 3134
#include "CUDA10_Registers_Kernel.h"
#define SALT 3135
#include "CUDA10_Registers_Kernel.h"
#define SALT 3136
#include "CUDA10_Registers_Kernel.h"
#define SALT 3137
#include "CUDA10_Registers_Kernel.h"
#define SALT 3138
#include "CUDA10_Registers_Kernel.h"
#define SALT 3139
#include "CUDA10_Registers_Kernel.h"
#define SALT 3140
#include "CUDA10_Registers_Kernel.h"
#define SALT 3141
#include "CUDA10_Registers_Kernel.h"
#define SALT 3142
#include "CUDA10_Registers_Kernel.h"
#define SALT 3143
#include "CUDA10_Registers_Kernel.h"
#define SALT 3144
#include "CUDA10_Registers_Kernel.h"
#define SALT 3145
#include "CUDA10_Registers_Kernel.h"
#define SALT 3146
#include "CUDA10_Registers_Kernel.h"
#define SALT 3147
#include "CUDA10_Registers_Kernel.h"
#define SALT 3148
#include "CUDA10_Registers_Kernel.h"
#define SALT 3149
#include "CUDA10_Registers_Kernel.h"
#define SALT 3150
#include "CUDA10_Registers_Kernel.h"
#define SALT 3151
#include "CUDA10_Registers_Kernel.h"
#define SALT 3152
#include "CUDA10_Registers_Kernel.h"
#define SALT 3153
#include "CUDA10_Registers_Kernel.h"
#define SALT 3154
#include "CUDA10_Registers_Kernel.h"
#define SALT 3155
#include "CUDA10_Registers_Kernel.h"
#define SALT 3156
#include "CUDA10_Registers_Kernel.h"
#define SALT 3157
#include "CUDA10_Registers_Kernel.h"
#define SALT 3158
#include "CUDA10_Registers_Kernel.h"
#define SALT 3159
#include "CUDA10_Registers_Kernel.h"
#define SALT 3160
#include "CUDA10_Registers_Kernel.h"
#define SALT 3161
#include "CUDA10_Registers_Kernel.h"
#define SALT 3162
#include "CUDA10_Registers_Kernel.h"
#define SALT 3163
#include "CUDA10_Registers_Kernel.h"
#define SALT 3164
#include "CUDA10_Registers_Kernel.h"
#define SALT 3165
#include "CUDA10_Registers_Kernel.h"
#define SALT 3166
#include "CUDA10_Registers_Kernel.h"
#define SALT 3167
#include "CUDA10_Registers_Kernel.h"
#define SALT 3168
#include "CUDA10_Registers_Kernel.h"
#define SALT 3169
#include "CUDA10_Registers_Kernel.h"
#define SALT 3170
#include "CUDA10_Registers_Kernel.h"
#define SALT 3171
#include "CUDA10_Registers_Kernel.h"
#define SALT 3172
#include "CUDA10_Registers_Kernel.h"
#define SALT 3173
#include "CUDA10_Registers_Kernel.h"
#define SALT 3174
#include "CUDA10_Registers_Kernel.h"
#define SALT 3175
#include "CUDA10_Registers_Kernel.h"
#define SALT 3176
#include "CUDA10_Registers_Kernel.h"
#define SALT 3177
#include "CUDA10_Registers_Kernel.h"
#define SALT 3178
#include "CUDA10_Registers_Kernel.h"
#define SALT 3179
#include "CUDA10_Registers_Kernel.h"
#define SALT 3180
#include "CUDA10_Registers_Kernel.h"
#define SALT 3181
#include "CUDA10_Registers_Kernel.h"
#define SALT 3182
#include "CUDA10_Registers_Kernel.h"
#define SALT 3183
#include "CUDA10_Registers_Kernel.h"
#define SALT 3184
#include "CUDA10_Registers_Kernel.h"
#define SALT 3185
#include "CUDA10_Registers_Kernel.h"
#define SALT 3186
#include "CUDA10_Registers_Kernel.h"
#define SALT 3187
#include "CUDA10_Registers_Kernel.h"
#define SALT 3188
#include "CUDA10_Registers_Kernel.h"
#define SALT 3189
#include "CUDA10_Registers_Kernel.h"
#define SALT 3190
#include "CUDA10_Registers_Kernel.h"
#define SALT 3191
#include "CUDA10_Registers_Kernel.h"
#define SALT 3192
#include "CUDA10_Registers_Kernel.h"
#define SALT 3193
#include "CUDA10_Registers_Kernel.h"
#define SALT 3194
#include "CUDA10_Registers_Kernel.h"
#define SALT 3195
#include "CUDA10_Registers_Kernel.h"
#define SALT 3196
#include "CUDA10_Registers_Kernel.h"
#define SALT 3197
#include "CUDA10_Registers_Kernel.h"
#define SALT 3198
#include "CUDA10_Registers_Kernel.h"
#define SALT 3199
#include "CUDA10_Registers_Kernel.h"
#define SALT 3200
#include "CUDA10_Registers_Kernel.h"
#define SALT 3201
#include "CUDA10_Registers_Kernel.h"
#define SALT 3202
#include "CUDA10_Registers_Kernel.h"
#define SALT 3203
#include "CUDA10_Registers_Kernel.h"
#define SALT 3204
#include "CUDA10_Registers_Kernel.h"
#define SALT 3205
#include "CUDA10_Registers_Kernel.h"
#define SALT 3206
#include "CUDA10_Registers_Kernel.h"
#define SALT 3207
#include "CUDA10_Registers_Kernel.h"
#define SALT 3208
#include "CUDA10_Registers_Kernel.h"
#define SALT 3209
#include "CUDA10_Registers_Kernel.h"
#define SALT 3210
#include "CUDA10_Registers_Kernel.h"
#define SALT 3211
#include "CUDA10_Registers_Kernel.h"
#define SALT 3212
#include "CUDA10_Registers_Kernel.h"
#define SALT 3213
#include "CUDA10_Registers_Kernel.h"
#define SALT 3214
#include "CUDA10_Registers_Kernel.h"
#define SALT 3215
#include "CUDA10_Registers_Kernel.h"
#define SALT 3216
#include "CUDA10_Registers_Kernel.h"
#define SALT 3217
#include "CUDA10_Registers_Kernel.h"
#define SALT 3218
#include "CUDA10_Registers_Kernel.h"
#define SALT 3219
#include "CUDA10_Registers_Kernel.h"
#define SALT 3220
#include "CUDA10_Registers_Kernel.h"
#define SALT 3221
#include "CUDA10_Registers_Kernel.h"
#define SALT 3222
#include "CUDA10_Registers_Kernel.h"
#define SALT 3223
#include "CUDA10_Registers_Kernel.h"
#define SALT 3224
#include "CUDA10_Registers_Kernel.h"
#define SALT 3225
#include "CUDA10_Registers_Kernel.h"
#define SALT 3226
#include "CUDA10_Registers_Kernel.h"
#define SALT 3227
#include "CUDA10_Registers_Kernel.h"
#define SALT 3228
#include "CUDA10_Registers_Kernel.h"
#define SALT 3229
#include "CUDA10_Registers_Kernel.h"
#define SALT 3230
#include "CUDA10_Registers_Kernel.h"
#define SALT 3231
#include "CUDA10_Registers_Kernel.h"
#define SALT 3232
#include "CUDA10_Registers_Kernel.h"
#define SALT 3233
#include "CUDA10_Registers_Kernel.h"
#define SALT 3234
#include "CUDA10_Registers_Kernel.h"
#define SALT 3235
#include "CUDA10_Registers_Kernel.h"
#define SALT 3236
#include "CUDA10_Registers_Kernel.h"
#define SALT 3237
#include "CUDA10_Registers_Kernel.h"
#define SALT 3238
#include "CUDA10_Registers_Kernel.h"
#define SALT 3239
#include "CUDA10_Registers_Kernel.h"
#define SALT 3240
#include "CUDA10_Registers_Kernel.h"
#define SALT 3241
#include "CUDA10_Registers_Kernel.h"
#define SALT 3242
#include "CUDA10_Registers_Kernel.h"
#define SALT 3243
#include "CUDA10_Registers_Kernel.h"
#define SALT 3244
#include "CUDA10_Registers_Kernel.h"
#define SALT 3245
#include "CUDA10_Registers_Kernel.h"
#define SALT 3246
#include "CUDA10_Registers_Kernel.h"
#define SALT 3247
#include "CUDA10_Registers_Kernel.h"
#define SALT 3248
#include "CUDA10_Registers_Kernel.h"
#define SALT 3249
#include "CUDA10_Registers_Kernel.h"
#define SALT 3250
#include "CUDA10_Registers_Kernel.h"
#define SALT 3251
#include "CUDA10_Registers_Kernel.h"
#define SALT 3252
#include "CUDA10_Registers_Kernel.h"
#define SALT 3253
#include "CUDA10_Registers_Kernel.h"
#define SALT 3254
#include "CUDA10_Registers_Kernel.h"
#define SALT 3255
#include "CUDA10_Registers_Kernel.h"
#define SALT 3256
#include "CUDA10_Registers_Kernel.h"
#define SALT 3257
#include "CUDA10_Registers_Kernel.h"
#define SALT 3258
#include "CUDA10_Registers_Kernel.h"
#define SALT 3259
#include "CUDA10_Registers_Kernel.h"
#define SALT 3260
#include "CUDA10_Registers_Kernel.h"
#define SALT 3261
#include "CUDA10_Registers_Kernel.h"
#define SALT 3262
#include "CUDA10_Registers_Kernel.h"
#define SALT 3263
#include "CUDA10_Registers_Kernel.h"
#define SALT 3264
#include "CUDA10_Registers_Kernel.h"
#define SALT 3265
#include "CUDA10_Registers_Kernel.h"
#define SALT 3266
#include "CUDA10_Registers_Kernel.h"
#define SALT 3267
#include "CUDA10_Registers_Kernel.h"
#define SALT 3268
#include "CUDA10_Registers_Kernel.h"
#define SALT 3269
#include "CUDA10_Registers_Kernel.h"
#define SALT 3270
#include "CUDA10_Registers_Kernel.h"
#define SALT 3271
#include "CUDA10_Registers_Kernel.h"
#define SALT 3272
#include "CUDA10_Registers_Kernel.h"
#define SALT 3273
#include "CUDA10_Registers_Kernel.h"
#define SALT 3274
#include "CUDA10_Registers_Kernel.h"
#define SALT 3275
#include "CUDA10_Registers_Kernel.h"
#define SALT 3276
#include "CUDA10_Registers_Kernel.h"
#define SALT 3277
#include "CUDA10_Registers_Kernel.h"
#define SALT 3278
#include "CUDA10_Registers_Kernel.h"
#define SALT 3279
#include "CUDA10_Registers_Kernel.h"
#define SALT 3280
#include "CUDA10_Registers_Kernel.h"
#define SALT 3281
#include "CUDA10_Registers_Kernel.h"
#define SALT 3282
#include "CUDA10_Registers_Kernel.h"
#define SALT 3283
#include "CUDA10_Registers_Kernel.h"
#define SALT 3284
#include "CUDA10_Registers_Kernel.h"
#define SALT 3285
#include "CUDA10_Registers_Kernel.h"
#define SALT 3286
#include "CUDA10_Registers_Kernel.h"
#define SALT 3287
#include "CUDA10_Registers_Kernel.h"
#define SALT 3288
#include "CUDA10_Registers_Kernel.h"
#define SALT 3289
#include "CUDA10_Registers_Kernel.h"
#define SALT 3290
#include "CUDA10_Registers_Kernel.h"
#define SALT 3291
#include "CUDA10_Registers_Kernel.h"
#define SALT 3292
#include "CUDA10_Registers_Kernel.h"
#define SALT 3293
#include "CUDA10_Registers_Kernel.h"
#define SALT 3294
#include "CUDA10_Registers_Kernel.h"
#define SALT 3295
#include "CUDA10_Registers_Kernel.h"
#define SALT 3296
#include "CUDA10_Registers_Kernel.h"
#define SALT 3297
#include "CUDA10_Registers_Kernel.h"
#define SALT 3298
#include "CUDA10_Registers_Kernel.h"
#define SALT 3299
#include "CUDA10_Registers_Kernel.h"
#define SALT 3300
#include "CUDA10_Registers_Kernel.h"
#define SALT 3301
#include "CUDA10_Registers_Kernel.h"
#define SALT 3302
#include "CUDA10_Registers_Kernel.h"
#define SALT 3303
#include "CUDA10_Registers_Kernel.h"
#define SALT 3304
#include "CUDA10_Registers_Kernel.h"
#define SALT 3305
#include "CUDA10_Registers_Kernel.h"
#define SALT 3306
#include "CUDA10_Registers_Kernel.h"
#define SALT 3307
#include "CUDA10_Registers_Kernel.h"
#define SALT 3308
#include "CUDA10_Registers_Kernel.h"
#define SALT 3309
#include "CUDA10_Registers_Kernel.h"
#define SALT 3310
#include "CUDA10_Registers_Kernel.h"
#define SALT 3311
#include "CUDA10_Registers_Kernel.h"
#define SALT 3312
#include "CUDA10_Registers_Kernel.h"
#define SALT 3313
#include "CUDA10_Registers_Kernel.h"
#define SALT 3314
#include "CUDA10_Registers_Kernel.h"
#define SALT 3315
#include "CUDA10_Registers_Kernel.h"
#define SALT 3316
#include "CUDA10_Registers_Kernel.h"
#define SALT 3317
#include "CUDA10_Registers_Kernel.h"
#define SALT 3318
#include "CUDA10_Registers_Kernel.h"
#define SALT 3319
#include "CUDA10_Registers_Kernel.h"
#define SALT 3320
#include "CUDA10_Registers_Kernel.h"
#define SALT 3321
#include "CUDA10_Registers_Kernel.h"
#define SALT 3322
#include "CUDA10_Registers_Kernel.h"
#define SALT 3323
#include "CUDA10_Registers_Kernel.h"
#define SALT 3324
#include "CUDA10_Registers_Kernel.h"
#define SALT 3325
#include "CUDA10_Registers_Kernel.h"
#define SALT 3326
#include "CUDA10_Registers_Kernel.h"
#define SALT 3327
#include "CUDA10_Registers_Kernel.h"
#define SALT 3328
#include "CUDA10_Registers_Kernel.h"
#define SALT 3329
#include "CUDA10_Registers_Kernel.h"
#define SALT 3330
#include "CUDA10_Registers_Kernel.h"
#define SALT 3331
#include "CUDA10_Registers_Kernel.h"
#define SALT 3332
#include "CUDA10_Registers_Kernel.h"
#define SALT 3333
#include "CUDA10_Registers_Kernel.h"
#define SALT 3334
#include "CUDA10_Registers_Kernel.h"
#define SALT 3335
#include "CUDA10_Registers_Kernel.h"
#define SALT 3336
#include "CUDA10_Registers_Kernel.h"
#define SALT 3337
#include "CUDA10_Registers_Kernel.h"
#define SALT 3338
#include "CUDA10_Registers_Kernel.h"
#define SALT 3339
#include "CUDA10_Registers_Kernel.h"
#define SALT 3340
#include "CUDA10_Registers_Kernel.h"
#define SALT 3341
#include "CUDA10_Registers_Kernel.h"
#define SALT 3342
#include "CUDA10_Registers_Kernel.h"
#define SALT 3343
#include "CUDA10_Registers_Kernel.h"
#define SALT 3344
#include "CUDA10_Registers_Kernel.h"
#define SALT 3345
#include "CUDA10_Registers_Kernel.h"
#define SALT 3346
#include "CUDA10_Registers_Kernel.h"
#define SALT 3347
#include "CUDA10_Registers_Kernel.h"
#define SALT 3348
#include "CUDA10_Registers_Kernel.h"
#define SALT 3349
#include "CUDA10_Registers_Kernel.h"
#define SALT 3350
#include "CUDA10_Registers_Kernel.h"
#define SALT 3351
#include "CUDA10_Registers_Kernel.h"
#define SALT 3352
#include "CUDA10_Registers_Kernel.h"
#define SALT 3353
#include "CUDA10_Registers_Kernel.h"
#define SALT 3354
#include "CUDA10_Registers_Kernel.h"
#define SALT 3355
#include "CUDA10_Registers_Kernel.h"
#define SALT 3356
#include "CUDA10_Registers_Kernel.h"
#define SALT 3357
#include "CUDA10_Registers_Kernel.h"
#define SALT 3358
#include "CUDA10_Registers_Kernel.h"
#define SALT 3359
#include "CUDA10_Registers_Kernel.h"
#define SALT 3360
#include "CUDA10_Registers_Kernel.h"
#define SALT 3361
#include "CUDA10_Registers_Kernel.h"
#define SALT 3362
#include "CUDA10_Registers_Kernel.h"
#define SALT 3363
#include "CUDA10_Registers_Kernel.h"
#define SALT 3364
#include "CUDA10_Registers_Kernel.h"
#define SALT 3365
#include "CUDA10_Registers_Kernel.h"
#define SALT 3366
#include "CUDA10_Registers_Kernel.h"
#define SALT 3367
#include "CUDA10_Registers_Kernel.h"
#define SALT 3368
#include "CUDA10_Registers_Kernel.h"
#define SALT 3369
#include "CUDA10_Registers_Kernel.h"
#define SALT 3370
#include "CUDA10_Registers_Kernel.h"
#define SALT 3371
#include "CUDA10_Registers_Kernel.h"
#define SALT 3372
#include "CUDA10_Registers_Kernel.h"
#define SALT 3373
#include "CUDA10_Registers_Kernel.h"
#define SALT 3374
#include "CUDA10_Registers_Kernel.h"
#define SALT 3375
#include "CUDA10_Registers_Kernel.h"
#define SALT 3376
#include "CUDA10_Registers_Kernel.h"
#define SALT 3377
#include "CUDA10_Registers_Kernel.h"
#define SALT 3378
#include "CUDA10_Registers_Kernel.h"
#define SALT 3379
#include "CUDA10_Registers_Kernel.h"
#define SALT 3380
#include "CUDA10_Registers_Kernel.h"
#define SALT 3381
#include "CUDA10_Registers_Kernel.h"
#define SALT 3382
#include "CUDA10_Registers_Kernel.h"
#define SALT 3383
#include "CUDA10_Registers_Kernel.h"
#define SALT 3384
#include "CUDA10_Registers_Kernel.h"
#define SALT 3385
#include "CUDA10_Registers_Kernel.h"
#define SALT 3386
#include "CUDA10_Registers_Kernel.h"
#define SALT 3387
#include "CUDA10_Registers_Kernel.h"
#define SALT 3388
#include "CUDA10_Registers_Kernel.h"
#define SALT 3389
#include "CUDA10_Registers_Kernel.h"
#define SALT 3390
#include "CUDA10_Registers_Kernel.h"
#define SALT 3391
#include "CUDA10_Registers_Kernel.h"
#define SALT 3392
#include "CUDA10_Registers_Kernel.h"
#define SALT 3393
#include "CUDA10_Registers_Kernel.h"
#define SALT 3394
#include "CUDA10_Registers_Kernel.h"
#define SALT 3395
#include "CUDA10_Registers_Kernel.h"
#define SALT 3396
#include "CUDA10_Registers_Kernel.h"
#define SALT 3397
#include "CUDA10_Registers_Kernel.h"
#define SALT 3398
#include "CUDA10_Registers_Kernel.h"
#define SALT 3399
#include "CUDA10_Registers_Kernel.h"
#define SALT 3400
#include "CUDA10_Registers_Kernel.h"
#define SALT 3401
#include "CUDA10_Registers_Kernel.h"
#define SALT 3402
#include "CUDA10_Registers_Kernel.h"
#define SALT 3403
#include "CUDA10_Registers_Kernel.h"
#define SALT 3404
#include "CUDA10_Registers_Kernel.h"
#define SALT 3405
#include "CUDA10_Registers_Kernel.h"
#define SALT 3406
#include "CUDA10_Registers_Kernel.h"
#define SALT 3407
#include "CUDA10_Registers_Kernel.h"
#define SALT 3408
#include "CUDA10_Registers_Kernel.h"
#define SALT 3409
#include "CUDA10_Registers_Kernel.h"
#define SALT 3410
#include "CUDA10_Registers_Kernel.h"
#define SALT 3411
#include "CUDA10_Registers_Kernel.h"
#define SALT 3412
#include "CUDA10_Registers_Kernel.h"
#define SALT 3413
#include "CUDA10_Registers_Kernel.h"
#define SALT 3414
#include "CUDA10_Registers_Kernel.h"
#define SALT 3415
#include "CUDA10_Registers_Kernel.h"
#define SALT 3416
#include "CUDA10_Registers_Kernel.h"
#define SALT 3417
#include "CUDA10_Registers_Kernel.h"
#define SALT 3418
#include "CUDA10_Registers_Kernel.h"
#define SALT 3419
#include "CUDA10_Registers_Kernel.h"
#define SALT 3420
#include "CUDA10_Registers_Kernel.h"
#define SALT 3421
#include "CUDA10_Registers_Kernel.h"
#define SALT 3422
#include "CUDA10_Registers_Kernel.h"
#define SALT 3423
#include "CUDA10_Registers_Kernel.h"
#define SALT 3424
#include "CUDA10_Registers_Kernel.h"
#define SALT 3425
#include "CUDA10_Registers_Kernel.h"
#define SALT 3426
#include "CUDA10_Registers_Kernel.h"
#define SALT 3427
#include "CUDA10_Registers_Kernel.h"
#define SALT 3428
#include "CUDA10_Registers_Kernel.h"
#define SALT 3429
#include "CUDA10_Registers_Kernel.h"
#define SALT 3430
#include "CUDA10_Registers_Kernel.h"
#define SALT 3431
#include "CUDA10_Registers_Kernel.h"
#define SALT 3432
#include "CUDA10_Registers_Kernel.h"
#define SALT 3433
#include "CUDA10_Registers_Kernel.h"
#define SALT 3434
#include "CUDA10_Registers_Kernel.h"
#define SALT 3435
#include "CUDA10_Registers_Kernel.h"
#define SALT 3436
#include "CUDA10_Registers_Kernel.h"
#define SALT 3437
#include "CUDA10_Registers_Kernel.h"
#define SALT 3438
#include "CUDA10_Registers_Kernel.h"
#define SALT 3439
#include "CUDA10_Registers_Kernel.h"
#define SALT 3440
#include "CUDA10_Registers_Kernel.h"
#define SALT 3441
#include "CUDA10_Registers_Kernel.h"
#define SALT 3442
#include "CUDA10_Registers_Kernel.h"
#define SALT 3443
#include "CUDA10_Registers_Kernel.h"
#define SALT 3444
#include "CUDA10_Registers_Kernel.h"
#define SALT 3445
#include "CUDA10_Registers_Kernel.h"
#define SALT 3446
#include "CUDA10_Registers_Kernel.h"
#define SALT 3447
#include "CUDA10_Registers_Kernel.h"
#define SALT 3448
#include "CUDA10_Registers_Kernel.h"
#define SALT 3449
#include "CUDA10_Registers_Kernel.h"
#define SALT 3450
#include "CUDA10_Registers_Kernel.h"
#define SALT 3451
#include "CUDA10_Registers_Kernel.h"
#define SALT 3452
#include "CUDA10_Registers_Kernel.h"
#define SALT 3453
#include "CUDA10_Registers_Kernel.h"
#define SALT 3454
#include "CUDA10_Registers_Kernel.h"
#define SALT 3455
#include "CUDA10_Registers_Kernel.h"
#define SALT 3456
#include "CUDA10_Registers_Kernel.h"
#define SALT 3457
#include "CUDA10_Registers_Kernel.h"
#define SALT 3458
#include "CUDA10_Registers_Kernel.h"
#define SALT 3459
#include "CUDA10_Registers_Kernel.h"
#define SALT 3460
#include "CUDA10_Registers_Kernel.h"
#define SALT 3461
#include "CUDA10_Registers_Kernel.h"
#define SALT 3462
#include "CUDA10_Registers_Kernel.h"
#define SALT 3463
#include "CUDA10_Registers_Kernel.h"
#define SALT 3464
#include "CUDA10_Registers_Kernel.h"
#define SALT 3465
#include "CUDA10_Registers_Kernel.h"
#define SALT 3466
#include "CUDA10_Registers_Kernel.h"
#define SALT 3467
#include "CUDA10_Registers_Kernel.h"
#define SALT 3468
#include "CUDA10_Registers_Kernel.h"
#define SALT 3469
#include "CUDA10_Registers_Kernel.h"
#define SALT 3470
#include "CUDA10_Registers_Kernel.h"
#define SALT 3471
#include "CUDA10_Registers_Kernel.h"
#define SALT 3472
#include "CUDA10_Registers_Kernel.h"
#define SALT 3473
#include "CUDA10_Registers_Kernel.h"
#define SALT 3474
#include "CUDA10_Registers_Kernel.h"
#define SALT 3475
#include "CUDA10_Registers_Kernel.h"
#define SALT 3476
#include "CUDA10_Registers_Kernel.h"
#define SALT 3477
#include "CUDA10_Registers_Kernel.h"
#define SALT 3478
#include "CUDA10_Registers_Kernel.h"
#define SALT 3479
#include "CUDA10_Registers_Kernel.h"
#define SALT 3480
#include "CUDA10_Registers_Kernel.h"
#define SALT 3481
#include "CUDA10_Registers_Kernel.h"
#define SALT 3482
#include "CUDA10_Registers_Kernel.h"
#define SALT 3483
#include "CUDA10_Registers_Kernel.h"
#define SALT 3484
#include "CUDA10_Registers_Kernel.h"
#define SALT 3485
#include "CUDA10_Registers_Kernel.h"
#define SALT 3486
#include "CUDA10_Registers_Kernel.h"
#define SALT 3487
#include "CUDA10_Registers_Kernel.h"
#define SALT 3488
#include "CUDA10_Registers_Kernel.h"
#define SALT 3489
#include "CUDA10_Registers_Kernel.h"
#define SALT 3490
#include "CUDA10_Registers_Kernel.h"
#define SALT 3491
#include "CUDA10_Registers_Kernel.h"
#define SALT 3492
#include "CUDA10_Registers_Kernel.h"
#define SALT 3493
#include "CUDA10_Registers_Kernel.h"
#define SALT 3494
#include "CUDA10_Registers_Kernel.h"
#define SALT 3495
#include "CUDA10_Registers_Kernel.h"
#define SALT 3496
#include "CUDA10_Registers_Kernel.h"
#define SALT 3497
#include "CUDA10_Registers_Kernel.h"
#define SALT 3498
#include "CUDA10_Registers_Kernel.h"
#define SALT 3499
#include "CUDA10_Registers_Kernel.h"
#define SALT 3500
#include "CUDA10_Registers_Kernel.h"
#define SALT 3501
#include "CUDA10_Registers_Kernel.h"
#define SALT 3502
#include "CUDA10_Registers_Kernel.h"
#define SALT 3503
#include "CUDA10_Registers_Kernel.h"
#define SALT 3504
#include "CUDA10_Registers_Kernel.h"
#define SALT 3505
#include "CUDA10_Registers_Kernel.h"
#define SALT 3506
#include "CUDA10_Registers_Kernel.h"
#define SALT 3507
#include "CUDA10_Registers_Kernel.h"
#define SALT 3508
#include "CUDA10_Registers_Kernel.h"
#define SALT 3509
#include "CUDA10_Registers_Kernel.h"
#define SALT 3510
#include "CUDA10_Registers_Kernel.h"
#define SALT 3511
#include "CUDA10_Registers_Kernel.h"
#define SALT 3512
#include "CUDA10_Registers_Kernel.h"
#define SALT 3513
#include "CUDA10_Registers_Kernel.h"
#define SALT 3514
#include "CUDA10_Registers_Kernel.h"
#define SALT 3515
#include "CUDA10_Registers_Kernel.h"
#define SALT 3516
#include "CUDA10_Registers_Kernel.h"
#define SALT 3517
#include "CUDA10_Registers_Kernel.h"
#define SALT 3518
#include "CUDA10_Registers_Kernel.h"
#define SALT 3519
#include "CUDA10_Registers_Kernel.h"
#define SALT 3520
#include "CUDA10_Registers_Kernel.h"
#define SALT 3521
#include "CUDA10_Registers_Kernel.h"
#define SALT 3522
#include "CUDA10_Registers_Kernel.h"
#define SALT 3523
#include "CUDA10_Registers_Kernel.h"
#define SALT 3524
#include "CUDA10_Registers_Kernel.h"
#define SALT 3525
#include "CUDA10_Registers_Kernel.h"
#define SALT 3526
#include "CUDA10_Registers_Kernel.h"
#define SALT 3527
#include "CUDA10_Registers_Kernel.h"
#define SALT 3528
#include "CUDA10_Registers_Kernel.h"
#define SALT 3529
#include "CUDA10_Registers_Kernel.h"
#define SALT 3530
#include "CUDA10_Registers_Kernel.h"
#define SALT 3531
#include "CUDA10_Registers_Kernel.h"
#define SALT 3532
#include "CUDA10_Registers_Kernel.h"
#define SALT 3533
#include "CUDA10_Registers_Kernel.h"
#define SALT 3534
#include "CUDA10_Registers_Kernel.h"
#define SALT 3535
#include "CUDA10_Registers_Kernel.h"
#define SALT 3536
#include "CUDA10_Registers_Kernel.h"
#define SALT 3537
#include "CUDA10_Registers_Kernel.h"
#define SALT 3538
#include "CUDA10_Registers_Kernel.h"
#define SALT 3539
#include "CUDA10_Registers_Kernel.h"
#define SALT 3540
#include "CUDA10_Registers_Kernel.h"
#define SALT 3541
#include "CUDA10_Registers_Kernel.h"
#define SALT 3542
#include "CUDA10_Registers_Kernel.h"
#define SALT 3543
#include "CUDA10_Registers_Kernel.h"
#define SALT 3544
#include "CUDA10_Registers_Kernel.h"
#define SALT 3545
#include "CUDA10_Registers_Kernel.h"
#define SALT 3546
#include "CUDA10_Registers_Kernel.h"
#define SALT 3547
#include "CUDA10_Registers_Kernel.h"
#define SALT 3548
#include "CUDA10_Registers_Kernel.h"
#define SALT 3549
#include "CUDA10_Registers_Kernel.h"
#define SALT 3550
#include "CUDA10_Registers_Kernel.h"
#define SALT 3551
#include "CUDA10_Registers_Kernel.h"
#define SALT 3552
#include "CUDA10_Registers_Kernel.h"
#define SALT 3553
#include "CUDA10_Registers_Kernel.h"
#define SALT 3554
#include "CUDA10_Registers_Kernel.h"
#define SALT 3555
#include "CUDA10_Registers_Kernel.h"
#define SALT 3556
#include "CUDA10_Registers_Kernel.h"
#define SALT 3557
#include "CUDA10_Registers_Kernel.h"
#define SALT 3558
#include "CUDA10_Registers_Kernel.h"
#define SALT 3559
#include "CUDA10_Registers_Kernel.h"
#define SALT 3560
#include "CUDA10_Registers_Kernel.h"
#define SALT 3561
#include "CUDA10_Registers_Kernel.h"
#define SALT 3562
#include "CUDA10_Registers_Kernel.h"
#define SALT 3563
#include "CUDA10_Registers_Kernel.h"
#define SALT 3564
#include "CUDA10_Registers_Kernel.h"
#define SALT 3565
#include "CUDA10_Registers_Kernel.h"
#define SALT 3566
#include "CUDA10_Registers_Kernel.h"
#define SALT 3567
#include "CUDA10_Registers_Kernel.h"
#define SALT 3568
#include "CUDA10_Registers_Kernel.h"
#define SALT 3569
#include "CUDA10_Registers_Kernel.h"
#define SALT 3570
#include "CUDA10_Registers_Kernel.h"
#define SALT 3571
#include "CUDA10_Registers_Kernel.h"
#define SALT 3572
#include "CUDA10_Registers_Kernel.h"
#define SALT 3573
#include "CUDA10_Registers_Kernel.h"
#define SALT 3574
#include "CUDA10_Registers_Kernel.h"
#define SALT 3575
#include "CUDA10_Registers_Kernel.h"
#define SALT 3576
#include "CUDA10_Registers_Kernel.h"
#define SALT 3577
#include "CUDA10_Registers_Kernel.h"
#define SALT 3578
#include "CUDA10_Registers_Kernel.h"
#define SALT 3579
#include "CUDA10_Registers_Kernel.h"
#define SALT 3580
#include "CUDA10_Registers_Kernel.h"
#define SALT 3581
#include "CUDA10_Registers_Kernel.h"
#define SALT 3582
#include "CUDA10_Registers_Kernel.h"
#define SALT 3583
#include "CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher6()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel6(
	unsigned int numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	unsigned int *cudaTripcodeChunkArray,
	unsigned int numTripcodeChunk,
	int intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 3072: LAUNCH_KERNEL(3072); break;
	case 3073: LAUNCH_KERNEL(3073); break;
	case 3074: LAUNCH_KERNEL(3074); break;
	case 3075: LAUNCH_KERNEL(3075); break;
	case 3076: LAUNCH_KERNEL(3076); break;
	case 3077: LAUNCH_KERNEL(3077); break;
	case 3078: LAUNCH_KERNEL(3078); break;
	case 3079: LAUNCH_KERNEL(3079); break;
	case 3080: LAUNCH_KERNEL(3080); break;
	case 3081: LAUNCH_KERNEL(3081); break;
	case 3082: LAUNCH_KERNEL(3082); break;
	case 3083: LAUNCH_KERNEL(3083); break;
	case 3084: LAUNCH_KERNEL(3084); break;
	case 3085: LAUNCH_KERNEL(3085); break;
	case 3086: LAUNCH_KERNEL(3086); break;
	case 3087: LAUNCH_KERNEL(3087); break;
	case 3088: LAUNCH_KERNEL(3088); break;
	case 3089: LAUNCH_KERNEL(3089); break;
	case 3090: LAUNCH_KERNEL(3090); break;
	case 3091: LAUNCH_KERNEL(3091); break;
	case 3092: LAUNCH_KERNEL(3092); break;
	case 3093: LAUNCH_KERNEL(3093); break;
	case 3094: LAUNCH_KERNEL(3094); break;
	case 3095: LAUNCH_KERNEL(3095); break;
	case 3096: LAUNCH_KERNEL(3096); break;
	case 3097: LAUNCH_KERNEL(3097); break;
	case 3098: LAUNCH_KERNEL(3098); break;
	case 3099: LAUNCH_KERNEL(3099); break;
	case 3100: LAUNCH_KERNEL(3100); break;
	case 3101: LAUNCH_KERNEL(3101); break;
	case 3102: LAUNCH_KERNEL(3102); break;
	case 3103: LAUNCH_KERNEL(3103); break;
	case 3104: LAUNCH_KERNEL(3104); break;
	case 3105: LAUNCH_KERNEL(3105); break;
	case 3106: LAUNCH_KERNEL(3106); break;
	case 3107: LAUNCH_KERNEL(3107); break;
	case 3108: LAUNCH_KERNEL(3108); break;
	case 3109: LAUNCH_KERNEL(3109); break;
	case 3110: LAUNCH_KERNEL(3110); break;
	case 3111: LAUNCH_KERNEL(3111); break;
	case 3112: LAUNCH_KERNEL(3112); break;
	case 3113: LAUNCH_KERNEL(3113); break;
	case 3114: LAUNCH_KERNEL(3114); break;
	case 3115: LAUNCH_KERNEL(3115); break;
	case 3116: LAUNCH_KERNEL(3116); break;
	case 3117: LAUNCH_KERNEL(3117); break;
	case 3118: LAUNCH_KERNEL(3118); break;
	case 3119: LAUNCH_KERNEL(3119); break;
	case 3120: LAUNCH_KERNEL(3120); break;
	case 3121: LAUNCH_KERNEL(3121); break;
	case 3122: LAUNCH_KERNEL(3122); break;
	case 3123: LAUNCH_KERNEL(3123); break;
	case 3124: LAUNCH_KERNEL(3124); break;
	case 3125: LAUNCH_KERNEL(3125); break;
	case 3126: LAUNCH_KERNEL(3126); break;
	case 3127: LAUNCH_KERNEL(3127); break;
	case 3128: LAUNCH_KERNEL(3128); break;
	case 3129: LAUNCH_KERNEL(3129); break;
	case 3130: LAUNCH_KERNEL(3130); break;
	case 3131: LAUNCH_KERNEL(3131); break;
	case 3132: LAUNCH_KERNEL(3132); break;
	case 3133: LAUNCH_KERNEL(3133); break;
	case 3134: LAUNCH_KERNEL(3134); break;
	case 3135: LAUNCH_KERNEL(3135); break;
	case 3136: LAUNCH_KERNEL(3136); break;
	case 3137: LAUNCH_KERNEL(3137); break;
	case 3138: LAUNCH_KERNEL(3138); break;
	case 3139: LAUNCH_KERNEL(3139); break;
	case 3140: LAUNCH_KERNEL(3140); break;
	case 3141: LAUNCH_KERNEL(3141); break;
	case 3142: LAUNCH_KERNEL(3142); break;
	case 3143: LAUNCH_KERNEL(3143); break;
	case 3144: LAUNCH_KERNEL(3144); break;
	case 3145: LAUNCH_KERNEL(3145); break;
	case 3146: LAUNCH_KERNEL(3146); break;
	case 3147: LAUNCH_KERNEL(3147); break;
	case 3148: LAUNCH_KERNEL(3148); break;
	case 3149: LAUNCH_KERNEL(3149); break;
	case 3150: LAUNCH_KERNEL(3150); break;
	case 3151: LAUNCH_KERNEL(3151); break;
	case 3152: LAUNCH_KERNEL(3152); break;
	case 3153: LAUNCH_KERNEL(3153); break;
	case 3154: LAUNCH_KERNEL(3154); break;
	case 3155: LAUNCH_KERNEL(3155); break;
	case 3156: LAUNCH_KERNEL(3156); break;
	case 3157: LAUNCH_KERNEL(3157); break;
	case 3158: LAUNCH_KERNEL(3158); break;
	case 3159: LAUNCH_KERNEL(3159); break;
	case 3160: LAUNCH_KERNEL(3160); break;
	case 3161: LAUNCH_KERNEL(3161); break;
	case 3162: LAUNCH_KERNEL(3162); break;
	case 3163: LAUNCH_KERNEL(3163); break;
	case 3164: LAUNCH_KERNEL(3164); break;
	case 3165: LAUNCH_KERNEL(3165); break;
	case 3166: LAUNCH_KERNEL(3166); break;
	case 3167: LAUNCH_KERNEL(3167); break;
	case 3168: LAUNCH_KERNEL(3168); break;
	case 3169: LAUNCH_KERNEL(3169); break;
	case 3170: LAUNCH_KERNEL(3170); break;
	case 3171: LAUNCH_KERNEL(3171); break;
	case 3172: LAUNCH_KERNEL(3172); break;
	case 3173: LAUNCH_KERNEL(3173); break;
	case 3174: LAUNCH_KERNEL(3174); break;
	case 3175: LAUNCH_KERNEL(3175); break;
	case 3176: LAUNCH_KERNEL(3176); break;
	case 3177: LAUNCH_KERNEL(3177); break;
	case 3178: LAUNCH_KERNEL(3178); break;
	case 3179: LAUNCH_KERNEL(3179); break;
	case 3180: LAUNCH_KERNEL(3180); break;
	case 3181: LAUNCH_KERNEL(3181); break;
	case 3182: LAUNCH_KERNEL(3182); break;
	case 3183: LAUNCH_KERNEL(3183); break;
	case 3184: LAUNCH_KERNEL(3184); break;
	case 3185: LAUNCH_KERNEL(3185); break;
	case 3186: LAUNCH_KERNEL(3186); break;
	case 3187: LAUNCH_KERNEL(3187); break;
	case 3188: LAUNCH_KERNEL(3188); break;
	case 3189: LAUNCH_KERNEL(3189); break;
	case 3190: LAUNCH_KERNEL(3190); break;
	case 3191: LAUNCH_KERNEL(3191); break;
	case 3192: LAUNCH_KERNEL(3192); break;
	case 3193: LAUNCH_KERNEL(3193); break;
	case 3194: LAUNCH_KERNEL(3194); break;
	case 3195: LAUNCH_KERNEL(3195); break;
	case 3196: LAUNCH_KERNEL(3196); break;
	case 3197: LAUNCH_KERNEL(3197); break;
	case 3198: LAUNCH_KERNEL(3198); break;
	case 3199: LAUNCH_KERNEL(3199); break;
	case 3200: LAUNCH_KERNEL(3200); break;
	case 3201: LAUNCH_KERNEL(3201); break;
	case 3202: LAUNCH_KERNEL(3202); break;
	case 3203: LAUNCH_KERNEL(3203); break;
	case 3204: LAUNCH_KERNEL(3204); break;
	case 3205: LAUNCH_KERNEL(3205); break;
	case 3206: LAUNCH_KERNEL(3206); break;
	case 3207: LAUNCH_KERNEL(3207); break;
	case 3208: LAUNCH_KERNEL(3208); break;
	case 3209: LAUNCH_KERNEL(3209); break;
	case 3210: LAUNCH_KERNEL(3210); break;
	case 3211: LAUNCH_KERNEL(3211); break;
	case 3212: LAUNCH_KERNEL(3212); break;
	case 3213: LAUNCH_KERNEL(3213); break;
	case 3214: LAUNCH_KERNEL(3214); break;
	case 3215: LAUNCH_KERNEL(3215); break;
	case 3216: LAUNCH_KERNEL(3216); break;
	case 3217: LAUNCH_KERNEL(3217); break;
	case 3218: LAUNCH_KERNEL(3218); break;
	case 3219: LAUNCH_KERNEL(3219); break;
	case 3220: LAUNCH_KERNEL(3220); break;
	case 3221: LAUNCH_KERNEL(3221); break;
	case 3222: LAUNCH_KERNEL(3222); break;
	case 3223: LAUNCH_KERNEL(3223); break;
	case 3224: LAUNCH_KERNEL(3224); break;
	case 3225: LAUNCH_KERNEL(3225); break;
	case 3226: LAUNCH_KERNEL(3226); break;
	case 3227: LAUNCH_KERNEL(3227); break;
	case 3228: LAUNCH_KERNEL(3228); break;
	case 3229: LAUNCH_KERNEL(3229); break;
	case 3230: LAUNCH_KERNEL(3230); break;
	case 3231: LAUNCH_KERNEL(3231); break;
	case 3232: LAUNCH_KERNEL(3232); break;
	case 3233: LAUNCH_KERNEL(3233); break;
	case 3234: LAUNCH_KERNEL(3234); break;
	case 3235: LAUNCH_KERNEL(3235); break;
	case 3236: LAUNCH_KERNEL(3236); break;
	case 3237: LAUNCH_KERNEL(3237); break;
	case 3238: LAUNCH_KERNEL(3238); break;
	case 3239: LAUNCH_KERNEL(3239); break;
	case 3240: LAUNCH_KERNEL(3240); break;
	case 3241: LAUNCH_KERNEL(3241); break;
	case 3242: LAUNCH_KERNEL(3242); break;
	case 3243: LAUNCH_KERNEL(3243); break;
	case 3244: LAUNCH_KERNEL(3244); break;
	case 3245: LAUNCH_KERNEL(3245); break;
	case 3246: LAUNCH_KERNEL(3246); break;
	case 3247: LAUNCH_KERNEL(3247); break;
	case 3248: LAUNCH_KERNEL(3248); break;
	case 3249: LAUNCH_KERNEL(3249); break;
	case 3250: LAUNCH_KERNEL(3250); break;
	case 3251: LAUNCH_KERNEL(3251); break;
	case 3252: LAUNCH_KERNEL(3252); break;
	case 3253: LAUNCH_KERNEL(3253); break;
	case 3254: LAUNCH_KERNEL(3254); break;
	case 3255: LAUNCH_KERNEL(3255); break;
	case 3256: LAUNCH_KERNEL(3256); break;
	case 3257: LAUNCH_KERNEL(3257); break;
	case 3258: LAUNCH_KERNEL(3258); break;
	case 3259: LAUNCH_KERNEL(3259); break;
	case 3260: LAUNCH_KERNEL(3260); break;
	case 3261: LAUNCH_KERNEL(3261); break;
	case 3262: LAUNCH_KERNEL(3262); break;
	case 3263: LAUNCH_KERNEL(3263); break;
	case 3264: LAUNCH_KERNEL(3264); break;
	case 3265: LAUNCH_KERNEL(3265); break;
	case 3266: LAUNCH_KERNEL(3266); break;
	case 3267: LAUNCH_KERNEL(3267); break;
	case 3268: LAUNCH_KERNEL(3268); break;
	case 3269: LAUNCH_KERNEL(3269); break;
	case 3270: LAUNCH_KERNEL(3270); break;
	case 3271: LAUNCH_KERNEL(3271); break;
	case 3272: LAUNCH_KERNEL(3272); break;
	case 3273: LAUNCH_KERNEL(3273); break;
	case 3274: LAUNCH_KERNEL(3274); break;
	case 3275: LAUNCH_KERNEL(3275); break;
	case 3276: LAUNCH_KERNEL(3276); break;
	case 3277: LAUNCH_KERNEL(3277); break;
	case 3278: LAUNCH_KERNEL(3278); break;
	case 3279: LAUNCH_KERNEL(3279); break;
	case 3280: LAUNCH_KERNEL(3280); break;
	case 3281: LAUNCH_KERNEL(3281); break;
	case 3282: LAUNCH_KERNEL(3282); break;
	case 3283: LAUNCH_KERNEL(3283); break;
	case 3284: LAUNCH_KERNEL(3284); break;
	case 3285: LAUNCH_KERNEL(3285); break;
	case 3286: LAUNCH_KERNEL(3286); break;
	case 3287: LAUNCH_KERNEL(3287); break;
	case 3288: LAUNCH_KERNEL(3288); break;
	case 3289: LAUNCH_KERNEL(3289); break;
	case 3290: LAUNCH_KERNEL(3290); break;
	case 3291: LAUNCH_KERNEL(3291); break;
	case 3292: LAUNCH_KERNEL(3292); break;
	case 3293: LAUNCH_KERNEL(3293); break;
	case 3294: LAUNCH_KERNEL(3294); break;
	case 3295: LAUNCH_KERNEL(3295); break;
	case 3296: LAUNCH_KERNEL(3296); break;
	case 3297: LAUNCH_KERNEL(3297); break;
	case 3298: LAUNCH_KERNEL(3298); break;
	case 3299: LAUNCH_KERNEL(3299); break;
	case 3300: LAUNCH_KERNEL(3300); break;
	case 3301: LAUNCH_KERNEL(3301); break;
	case 3302: LAUNCH_KERNEL(3302); break;
	case 3303: LAUNCH_KERNEL(3303); break;
	case 3304: LAUNCH_KERNEL(3304); break;
	case 3305: LAUNCH_KERNEL(3305); break;
	case 3306: LAUNCH_KERNEL(3306); break;
	case 3307: LAUNCH_KERNEL(3307); break;
	case 3308: LAUNCH_KERNEL(3308); break;
	case 3309: LAUNCH_KERNEL(3309); break;
	case 3310: LAUNCH_KERNEL(3310); break;
	case 3311: LAUNCH_KERNEL(3311); break;
	case 3312: LAUNCH_KERNEL(3312); break;
	case 3313: LAUNCH_KERNEL(3313); break;
	case 3314: LAUNCH_KERNEL(3314); break;
	case 3315: LAUNCH_KERNEL(3315); break;
	case 3316: LAUNCH_KERNEL(3316); break;
	case 3317: LAUNCH_KERNEL(3317); break;
	case 3318: LAUNCH_KERNEL(3318); break;
	case 3319: LAUNCH_KERNEL(3319); break;
	case 3320: LAUNCH_KERNEL(3320); break;
	case 3321: LAUNCH_KERNEL(3321); break;
	case 3322: LAUNCH_KERNEL(3322); break;
	case 3323: LAUNCH_KERNEL(3323); break;
	case 3324: LAUNCH_KERNEL(3324); break;
	case 3325: LAUNCH_KERNEL(3325); break;
	case 3326: LAUNCH_KERNEL(3326); break;
	case 3327: LAUNCH_KERNEL(3327); break;
	case 3328: LAUNCH_KERNEL(3328); break;
	case 3329: LAUNCH_KERNEL(3329); break;
	case 3330: LAUNCH_KERNEL(3330); break;
	case 3331: LAUNCH_KERNEL(3331); break;
	case 3332: LAUNCH_KERNEL(3332); break;
	case 3333: LAUNCH_KERNEL(3333); break;
	case 3334: LAUNCH_KERNEL(3334); break;
	case 3335: LAUNCH_KERNEL(3335); break;
	case 3336: LAUNCH_KERNEL(3336); break;
	case 3337: LAUNCH_KERNEL(3337); break;
	case 3338: LAUNCH_KERNEL(3338); break;
	case 3339: LAUNCH_KERNEL(3339); break;
	case 3340: LAUNCH_KERNEL(3340); break;
	case 3341: LAUNCH_KERNEL(3341); break;
	case 3342: LAUNCH_KERNEL(3342); break;
	case 3343: LAUNCH_KERNEL(3343); break;
	case 3344: LAUNCH_KERNEL(3344); break;
	case 3345: LAUNCH_KERNEL(3345); break;
	case 3346: LAUNCH_KERNEL(3346); break;
	case 3347: LAUNCH_KERNEL(3347); break;
	case 3348: LAUNCH_KERNEL(3348); break;
	case 3349: LAUNCH_KERNEL(3349); break;
	case 3350: LAUNCH_KERNEL(3350); break;
	case 3351: LAUNCH_KERNEL(3351); break;
	case 3352: LAUNCH_KERNEL(3352); break;
	case 3353: LAUNCH_KERNEL(3353); break;
	case 3354: LAUNCH_KERNEL(3354); break;
	case 3355: LAUNCH_KERNEL(3355); break;
	case 3356: LAUNCH_KERNEL(3356); break;
	case 3357: LAUNCH_KERNEL(3357); break;
	case 3358: LAUNCH_KERNEL(3358); break;
	case 3359: LAUNCH_KERNEL(3359); break;
	case 3360: LAUNCH_KERNEL(3360); break;
	case 3361: LAUNCH_KERNEL(3361); break;
	case 3362: LAUNCH_KERNEL(3362); break;
	case 3363: LAUNCH_KERNEL(3363); break;
	case 3364: LAUNCH_KERNEL(3364); break;
	case 3365: LAUNCH_KERNEL(3365); break;
	case 3366: LAUNCH_KERNEL(3366); break;
	case 3367: LAUNCH_KERNEL(3367); break;
	case 3368: LAUNCH_KERNEL(3368); break;
	case 3369: LAUNCH_KERNEL(3369); break;
	case 3370: LAUNCH_KERNEL(3370); break;
	case 3371: LAUNCH_KERNEL(3371); break;
	case 3372: LAUNCH_KERNEL(3372); break;
	case 3373: LAUNCH_KERNEL(3373); break;
	case 3374: LAUNCH_KERNEL(3374); break;
	case 3375: LAUNCH_KERNEL(3375); break;
	case 3376: LAUNCH_KERNEL(3376); break;
	case 3377: LAUNCH_KERNEL(3377); break;
	case 3378: LAUNCH_KERNEL(3378); break;
	case 3379: LAUNCH_KERNEL(3379); break;
	case 3380: LAUNCH_KERNEL(3380); break;
	case 3381: LAUNCH_KERNEL(3381); break;
	case 3382: LAUNCH_KERNEL(3382); break;
	case 3383: LAUNCH_KERNEL(3383); break;
	case 3384: LAUNCH_KERNEL(3384); break;
	case 3385: LAUNCH_KERNEL(3385); break;
	case 3386: LAUNCH_KERNEL(3386); break;
	case 3387: LAUNCH_KERNEL(3387); break;
	case 3388: LAUNCH_KERNEL(3388); break;
	case 3389: LAUNCH_KERNEL(3389); break;
	case 3390: LAUNCH_KERNEL(3390); break;
	case 3391: LAUNCH_KERNEL(3391); break;
	case 3392: LAUNCH_KERNEL(3392); break;
	case 3393: LAUNCH_KERNEL(3393); break;
	case 3394: LAUNCH_KERNEL(3394); break;
	case 3395: LAUNCH_KERNEL(3395); break;
	case 3396: LAUNCH_KERNEL(3396); break;
	case 3397: LAUNCH_KERNEL(3397); break;
	case 3398: LAUNCH_KERNEL(3398); break;
	case 3399: LAUNCH_KERNEL(3399); break;
	case 3400: LAUNCH_KERNEL(3400); break;
	case 3401: LAUNCH_KERNEL(3401); break;
	case 3402: LAUNCH_KERNEL(3402); break;
	case 3403: LAUNCH_KERNEL(3403); break;
	case 3404: LAUNCH_KERNEL(3404); break;
	case 3405: LAUNCH_KERNEL(3405); break;
	case 3406: LAUNCH_KERNEL(3406); break;
	case 3407: LAUNCH_KERNEL(3407); break;
	case 3408: LAUNCH_KERNEL(3408); break;
	case 3409: LAUNCH_KERNEL(3409); break;
	case 3410: LAUNCH_KERNEL(3410); break;
	case 3411: LAUNCH_KERNEL(3411); break;
	case 3412: LAUNCH_KERNEL(3412); break;
	case 3413: LAUNCH_KERNEL(3413); break;
	case 3414: LAUNCH_KERNEL(3414); break;
	case 3415: LAUNCH_KERNEL(3415); break;
	case 3416: LAUNCH_KERNEL(3416); break;
	case 3417: LAUNCH_KERNEL(3417); break;
	case 3418: LAUNCH_KERNEL(3418); break;
	case 3419: LAUNCH_KERNEL(3419); break;
	case 3420: LAUNCH_KERNEL(3420); break;
	case 3421: LAUNCH_KERNEL(3421); break;
	case 3422: LAUNCH_KERNEL(3422); break;
	case 3423: LAUNCH_KERNEL(3423); break;
	case 3424: LAUNCH_KERNEL(3424); break;
	case 3425: LAUNCH_KERNEL(3425); break;
	case 3426: LAUNCH_KERNEL(3426); break;
	case 3427: LAUNCH_KERNEL(3427); break;
	case 3428: LAUNCH_KERNEL(3428); break;
	case 3429: LAUNCH_KERNEL(3429); break;
	case 3430: LAUNCH_KERNEL(3430); break;
	case 3431: LAUNCH_KERNEL(3431); break;
	case 3432: LAUNCH_KERNEL(3432); break;
	case 3433: LAUNCH_KERNEL(3433); break;
	case 3434: LAUNCH_KERNEL(3434); break;
	case 3435: LAUNCH_KERNEL(3435); break;
	case 3436: LAUNCH_KERNEL(3436); break;
	case 3437: LAUNCH_KERNEL(3437); break;
	case 3438: LAUNCH_KERNEL(3438); break;
	case 3439: LAUNCH_KERNEL(3439); break;
	case 3440: LAUNCH_KERNEL(3440); break;
	case 3441: LAUNCH_KERNEL(3441); break;
	case 3442: LAUNCH_KERNEL(3442); break;
	case 3443: LAUNCH_KERNEL(3443); break;
	case 3444: LAUNCH_KERNEL(3444); break;
	case 3445: LAUNCH_KERNEL(3445); break;
	case 3446: LAUNCH_KERNEL(3446); break;
	case 3447: LAUNCH_KERNEL(3447); break;
	case 3448: LAUNCH_KERNEL(3448); break;
	case 3449: LAUNCH_KERNEL(3449); break;
	case 3450: LAUNCH_KERNEL(3450); break;
	case 3451: LAUNCH_KERNEL(3451); break;
	case 3452: LAUNCH_KERNEL(3452); break;
	case 3453: LAUNCH_KERNEL(3453); break;
	case 3454: LAUNCH_KERNEL(3454); break;
	case 3455: LAUNCH_KERNEL(3455); break;
	case 3456: LAUNCH_KERNEL(3456); break;
	case 3457: LAUNCH_KERNEL(3457); break;
	case 3458: LAUNCH_KERNEL(3458); break;
	case 3459: LAUNCH_KERNEL(3459); break;
	case 3460: LAUNCH_KERNEL(3460); break;
	case 3461: LAUNCH_KERNEL(3461); break;
	case 3462: LAUNCH_KERNEL(3462); break;
	case 3463: LAUNCH_KERNEL(3463); break;
	case 3464: LAUNCH_KERNEL(3464); break;
	case 3465: LAUNCH_KERNEL(3465); break;
	case 3466: LAUNCH_KERNEL(3466); break;
	case 3467: LAUNCH_KERNEL(3467); break;
	case 3468: LAUNCH_KERNEL(3468); break;
	case 3469: LAUNCH_KERNEL(3469); break;
	case 3470: LAUNCH_KERNEL(3470); break;
	case 3471: LAUNCH_KERNEL(3471); break;
	case 3472: LAUNCH_KERNEL(3472); break;
	case 3473: LAUNCH_KERNEL(3473); break;
	case 3474: LAUNCH_KERNEL(3474); break;
	case 3475: LAUNCH_KERNEL(3475); break;
	case 3476: LAUNCH_KERNEL(3476); break;
	case 3477: LAUNCH_KERNEL(3477); break;
	case 3478: LAUNCH_KERNEL(3478); break;
	case 3479: LAUNCH_KERNEL(3479); break;
	case 3480: LAUNCH_KERNEL(3480); break;
	case 3481: LAUNCH_KERNEL(3481); break;
	case 3482: LAUNCH_KERNEL(3482); break;
	case 3483: LAUNCH_KERNEL(3483); break;
	case 3484: LAUNCH_KERNEL(3484); break;
	case 3485: LAUNCH_KERNEL(3485); break;
	case 3486: LAUNCH_KERNEL(3486); break;
	case 3487: LAUNCH_KERNEL(3487); break;
	case 3488: LAUNCH_KERNEL(3488); break;
	case 3489: LAUNCH_KERNEL(3489); break;
	case 3490: LAUNCH_KERNEL(3490); break;
	case 3491: LAUNCH_KERNEL(3491); break;
	case 3492: LAUNCH_KERNEL(3492); break;
	case 3493: LAUNCH_KERNEL(3493); break;
	case 3494: LAUNCH_KERNEL(3494); break;
	case 3495: LAUNCH_KERNEL(3495); break;
	case 3496: LAUNCH_KERNEL(3496); break;
	case 3497: LAUNCH_KERNEL(3497); break;
	case 3498: LAUNCH_KERNEL(3498); break;
	case 3499: LAUNCH_KERNEL(3499); break;
	case 3500: LAUNCH_KERNEL(3500); break;
	case 3501: LAUNCH_KERNEL(3501); break;
	case 3502: LAUNCH_KERNEL(3502); break;
	case 3503: LAUNCH_KERNEL(3503); break;
	case 3504: LAUNCH_KERNEL(3504); break;
	case 3505: LAUNCH_KERNEL(3505); break;
	case 3506: LAUNCH_KERNEL(3506); break;
	case 3507: LAUNCH_KERNEL(3507); break;
	case 3508: LAUNCH_KERNEL(3508); break;
	case 3509: LAUNCH_KERNEL(3509); break;
	case 3510: LAUNCH_KERNEL(3510); break;
	case 3511: LAUNCH_KERNEL(3511); break;
	case 3512: LAUNCH_KERNEL(3512); break;
	case 3513: LAUNCH_KERNEL(3513); break;
	case 3514: LAUNCH_KERNEL(3514); break;
	case 3515: LAUNCH_KERNEL(3515); break;
	case 3516: LAUNCH_KERNEL(3516); break;
	case 3517: LAUNCH_KERNEL(3517); break;
	case 3518: LAUNCH_KERNEL(3518); break;
	case 3519: LAUNCH_KERNEL(3519); break;
	case 3520: LAUNCH_KERNEL(3520); break;
	case 3521: LAUNCH_KERNEL(3521); break;
	case 3522: LAUNCH_KERNEL(3522); break;
	case 3523: LAUNCH_KERNEL(3523); break;
	case 3524: LAUNCH_KERNEL(3524); break;
	case 3525: LAUNCH_KERNEL(3525); break;
	case 3526: LAUNCH_KERNEL(3526); break;
	case 3527: LAUNCH_KERNEL(3527); break;
	case 3528: LAUNCH_KERNEL(3528); break;
	case 3529: LAUNCH_KERNEL(3529); break;
	case 3530: LAUNCH_KERNEL(3530); break;
	case 3531: LAUNCH_KERNEL(3531); break;
	case 3532: LAUNCH_KERNEL(3532); break;
	case 3533: LAUNCH_KERNEL(3533); break;
	case 3534: LAUNCH_KERNEL(3534); break;
	case 3535: LAUNCH_KERNEL(3535); break;
	case 3536: LAUNCH_KERNEL(3536); break;
	case 3537: LAUNCH_KERNEL(3537); break;
	case 3538: LAUNCH_KERNEL(3538); break;
	case 3539: LAUNCH_KERNEL(3539); break;
	case 3540: LAUNCH_KERNEL(3540); break;
	case 3541: LAUNCH_KERNEL(3541); break;
	case 3542: LAUNCH_KERNEL(3542); break;
	case 3543: LAUNCH_KERNEL(3543); break;
	case 3544: LAUNCH_KERNEL(3544); break;
	case 3545: LAUNCH_KERNEL(3545); break;
	case 3546: LAUNCH_KERNEL(3546); break;
	case 3547: LAUNCH_KERNEL(3547); break;
	case 3548: LAUNCH_KERNEL(3548); break;
	case 3549: LAUNCH_KERNEL(3549); break;
	case 3550: LAUNCH_KERNEL(3550); break;
	case 3551: LAUNCH_KERNEL(3551); break;
	case 3552: LAUNCH_KERNEL(3552); break;
	case 3553: LAUNCH_KERNEL(3553); break;
	case 3554: LAUNCH_KERNEL(3554); break;
	case 3555: LAUNCH_KERNEL(3555); break;
	case 3556: LAUNCH_KERNEL(3556); break;
	case 3557: LAUNCH_KERNEL(3557); break;
	case 3558: LAUNCH_KERNEL(3558); break;
	case 3559: LAUNCH_KERNEL(3559); break;
	case 3560: LAUNCH_KERNEL(3560); break;
	case 3561: LAUNCH_KERNEL(3561); break;
	case 3562: LAUNCH_KERNEL(3562); break;
	case 3563: LAUNCH_KERNEL(3563); break;
	case 3564: LAUNCH_KERNEL(3564); break;
	case 3565: LAUNCH_KERNEL(3565); break;
	case 3566: LAUNCH_KERNEL(3566); break;
	case 3567: LAUNCH_KERNEL(3567); break;
	case 3568: LAUNCH_KERNEL(3568); break;
	case 3569: LAUNCH_KERNEL(3569); break;
	case 3570: LAUNCH_KERNEL(3570); break;
	case 3571: LAUNCH_KERNEL(3571); break;
	case 3572: LAUNCH_KERNEL(3572); break;
	case 3573: LAUNCH_KERNEL(3573); break;
	case 3574: LAUNCH_KERNEL(3574); break;
	case 3575: LAUNCH_KERNEL(3575); break;
	case 3576: LAUNCH_KERNEL(3576); break;
	case 3577: LAUNCH_KERNEL(3577); break;
	case 3578: LAUNCH_KERNEL(3578); break;
	case 3579: LAUNCH_KERNEL(3579); break;
	case 3580: LAUNCH_KERNEL(3580); break;
	case 3581: LAUNCH_KERNEL(3581); break;
	case 3582: LAUNCH_KERNEL(3582); break;
	case 3583: LAUNCH_KERNEL(3583); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
