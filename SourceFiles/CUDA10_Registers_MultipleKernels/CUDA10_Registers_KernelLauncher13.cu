// Meriken's Tripcode Engine
// Copyright (c) 2011-2016 /Meriken/. <meriken.ygch.net@gmail.com>
//
// The initial versions of this software were based on:
// CUDA SHA-1 Tripper 0.2.1
// Copyright (c) 2009 Horo/.IBXjcg
// 
// The code that deals with DES decryption is partially adopted from:
// John the Ripper password cracker
// Copyright (c) 1996-2002, 2005, 2010 by Solar Designer
// DeepLearningJohnDoe's fork of Meriken's Tripcode Engine
// Copyright (c) 2015 by <deeplearningjohndoe at gmail.com>
//
// The code that deals with SHA-1 hash generation is partially adopted from:
// sha_digest-2.2
// Copyright (C) 2009 Jens Thoms Toerring <jt@toerring.de>
// VecTripper 
// Copyright (C) 2011 tmkk <tmkk@smoug.net>
// 
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.



#include "../MerikensTripcodeEngine.h"

#ifdef CUDA_DES_ENABLE_MULTIPLE_KERNELS_MODE

#include "../CUDA10_Registers_Kernel_Common.h"

#define SALT 3328
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3329
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3330
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3331
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3332
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3333
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3334
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3335
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3336
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3337
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3338
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3339
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3340
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3341
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3342
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3343
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3344
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3345
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3346
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3347
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3348
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3349
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3350
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3351
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3352
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3353
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3354
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3355
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3356
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3357
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3358
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3359
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3360
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3361
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3362
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3363
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3364
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3365
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3366
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3367
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3368
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3369
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3370
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3371
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3372
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3373
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3374
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3375
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3376
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3377
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3378
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3379
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3380
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3381
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3382
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3383
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3384
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3385
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3386
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3387
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3388
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3389
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3390
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3391
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3392
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3393
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3394
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3395
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3396
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3397
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3398
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3399
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3400
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3401
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3402
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3403
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3404
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3405
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3406
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3407
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3408
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3409
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3410
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3411
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3412
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3413
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3414
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3415
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3416
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3417
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3418
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3419
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3420
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3421
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3422
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3423
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3424
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3425
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3426
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3427
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3428
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3429
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3430
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3431
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3432
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3433
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3434
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3435
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3436
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3437
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3438
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3439
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3440
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3441
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3442
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3443
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3444
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3445
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3446
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3447
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3448
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3449
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3450
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3451
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3452
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3453
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3454
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3455
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3456
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3457
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3458
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3459
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3460
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3461
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3462
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3463
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3464
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3465
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3466
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3467
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3468
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3469
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3470
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3471
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3472
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3473
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3474
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3475
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3476
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3477
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3478
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3479
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3480
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3481
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3482
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3483
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3484
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3485
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3486
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3487
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3488
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3489
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3490
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3491
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3492
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3493
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3494
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3495
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3496
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3497
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3498
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3499
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3500
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3501
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3502
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3503
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3504
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3505
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3506
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3507
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3508
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3509
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3510
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3511
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3512
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3513
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3514
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3515
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3516
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3517
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3518
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3519
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3520
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3521
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3522
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3523
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3524
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3525
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3526
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3527
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3528
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3529
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3530
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3531
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3532
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3533
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3534
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3535
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3536
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3537
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3538
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3539
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3540
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3541
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3542
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3543
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3544
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3545
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3546
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3547
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3548
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3549
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3550
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3551
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3552
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3553
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3554
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3555
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3556
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3557
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3558
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3559
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3560
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3561
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3562
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3563
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3564
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3565
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3566
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3567
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3568
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3569
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3570
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3571
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3572
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3573
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3574
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3575
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3576
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3577
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3578
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3579
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3580
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3581
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3582
#include "../CUDA10_Registers_Kernel.h"
#define SALT 3583
#include "../CUDA10_Registers_Kernel.h"



void CUDA_DES_InitializeKernelLauncher13()
{
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_FirstByte),   keyCharTable_FirstByte,   SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaKeyCharTable_SecondByte),  keyCharTable_SecondByte,  SIZE_KEY_CHAR_TABLE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaChunkBitmap),               chunkBitmap,               CHUNK_BITMAP_SIZE));
	CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cudaCompactMediumChunkBitmap),    compactMediumChunkBitmap,  COMPACT_MEDIUM_CHUNK_BITMAP_SIZE));
}

void CUDA_DES_LaunchKernel13(
	uint32_t numBlocksPerGrid,
	hipDeviceProp_t CUDADeviceProperties,
	hipStream_t currentStream,
	unsigned char *cudaPassCountArray,
	unsigned char *cudaTripcodeIndexArray,
	uint32_t *cudaTripcodeChunkArray,
	uint32_t numTripcodeChunk,
	int32_t intSalt,
	unsigned char *cudaKey0Array,
	unsigned char *cudaKey7Array,
	DES_Vector *cudaKeyVectorsFrom49To55,
	unsigned char *cudaKeyAndRandomBytes,
	int32_t searchMode)
{
	dim3 dimGrid(numBlocksPerGrid);
	dim3 dimBlock(CUDA_DES_NUM_THREADS_PER_BLOCK);
	switch (intSalt) {
	case 3328: LAUNCH_KERNEL(3328); break;
	case 3329: LAUNCH_KERNEL(3329); break;
	case 3330: LAUNCH_KERNEL(3330); break;
	case 3331: LAUNCH_KERNEL(3331); break;
	case 3332: LAUNCH_KERNEL(3332); break;
	case 3333: LAUNCH_KERNEL(3333); break;
	case 3334: LAUNCH_KERNEL(3334); break;
	case 3335: LAUNCH_KERNEL(3335); break;
	case 3336: LAUNCH_KERNEL(3336); break;
	case 3337: LAUNCH_KERNEL(3337); break;
	case 3338: LAUNCH_KERNEL(3338); break;
	case 3339: LAUNCH_KERNEL(3339); break;
	case 3340: LAUNCH_KERNEL(3340); break;
	case 3341: LAUNCH_KERNEL(3341); break;
	case 3342: LAUNCH_KERNEL(3342); break;
	case 3343: LAUNCH_KERNEL(3343); break;
	case 3344: LAUNCH_KERNEL(3344); break;
	case 3345: LAUNCH_KERNEL(3345); break;
	case 3346: LAUNCH_KERNEL(3346); break;
	case 3347: LAUNCH_KERNEL(3347); break;
	case 3348: LAUNCH_KERNEL(3348); break;
	case 3349: LAUNCH_KERNEL(3349); break;
	case 3350: LAUNCH_KERNEL(3350); break;
	case 3351: LAUNCH_KERNEL(3351); break;
	case 3352: LAUNCH_KERNEL(3352); break;
	case 3353: LAUNCH_KERNEL(3353); break;
	case 3354: LAUNCH_KERNEL(3354); break;
	case 3355: LAUNCH_KERNEL(3355); break;
	case 3356: LAUNCH_KERNEL(3356); break;
	case 3357: LAUNCH_KERNEL(3357); break;
	case 3358: LAUNCH_KERNEL(3358); break;
	case 3359: LAUNCH_KERNEL(3359); break;
	case 3360: LAUNCH_KERNEL(3360); break;
	case 3361: LAUNCH_KERNEL(3361); break;
	case 3362: LAUNCH_KERNEL(3362); break;
	case 3363: LAUNCH_KERNEL(3363); break;
	case 3364: LAUNCH_KERNEL(3364); break;
	case 3365: LAUNCH_KERNEL(3365); break;
	case 3366: LAUNCH_KERNEL(3366); break;
	case 3367: LAUNCH_KERNEL(3367); break;
	case 3368: LAUNCH_KERNEL(3368); break;
	case 3369: LAUNCH_KERNEL(3369); break;
	case 3370: LAUNCH_KERNEL(3370); break;
	case 3371: LAUNCH_KERNEL(3371); break;
	case 3372: LAUNCH_KERNEL(3372); break;
	case 3373: LAUNCH_KERNEL(3373); break;
	case 3374: LAUNCH_KERNEL(3374); break;
	case 3375: LAUNCH_KERNEL(3375); break;
	case 3376: LAUNCH_KERNEL(3376); break;
	case 3377: LAUNCH_KERNEL(3377); break;
	case 3378: LAUNCH_KERNEL(3378); break;
	case 3379: LAUNCH_KERNEL(3379); break;
	case 3380: LAUNCH_KERNEL(3380); break;
	case 3381: LAUNCH_KERNEL(3381); break;
	case 3382: LAUNCH_KERNEL(3382); break;
	case 3383: LAUNCH_KERNEL(3383); break;
	case 3384: LAUNCH_KERNEL(3384); break;
	case 3385: LAUNCH_KERNEL(3385); break;
	case 3386: LAUNCH_KERNEL(3386); break;
	case 3387: LAUNCH_KERNEL(3387); break;
	case 3388: LAUNCH_KERNEL(3388); break;
	case 3389: LAUNCH_KERNEL(3389); break;
	case 3390: LAUNCH_KERNEL(3390); break;
	case 3391: LAUNCH_KERNEL(3391); break;
	case 3392: LAUNCH_KERNEL(3392); break;
	case 3393: LAUNCH_KERNEL(3393); break;
	case 3394: LAUNCH_KERNEL(3394); break;
	case 3395: LAUNCH_KERNEL(3395); break;
	case 3396: LAUNCH_KERNEL(3396); break;
	case 3397: LAUNCH_KERNEL(3397); break;
	case 3398: LAUNCH_KERNEL(3398); break;
	case 3399: LAUNCH_KERNEL(3399); break;
	case 3400: LAUNCH_KERNEL(3400); break;
	case 3401: LAUNCH_KERNEL(3401); break;
	case 3402: LAUNCH_KERNEL(3402); break;
	case 3403: LAUNCH_KERNEL(3403); break;
	case 3404: LAUNCH_KERNEL(3404); break;
	case 3405: LAUNCH_KERNEL(3405); break;
	case 3406: LAUNCH_KERNEL(3406); break;
	case 3407: LAUNCH_KERNEL(3407); break;
	case 3408: LAUNCH_KERNEL(3408); break;
	case 3409: LAUNCH_KERNEL(3409); break;
	case 3410: LAUNCH_KERNEL(3410); break;
	case 3411: LAUNCH_KERNEL(3411); break;
	case 3412: LAUNCH_KERNEL(3412); break;
	case 3413: LAUNCH_KERNEL(3413); break;
	case 3414: LAUNCH_KERNEL(3414); break;
	case 3415: LAUNCH_KERNEL(3415); break;
	case 3416: LAUNCH_KERNEL(3416); break;
	case 3417: LAUNCH_KERNEL(3417); break;
	case 3418: LAUNCH_KERNEL(3418); break;
	case 3419: LAUNCH_KERNEL(3419); break;
	case 3420: LAUNCH_KERNEL(3420); break;
	case 3421: LAUNCH_KERNEL(3421); break;
	case 3422: LAUNCH_KERNEL(3422); break;
	case 3423: LAUNCH_KERNEL(3423); break;
	case 3424: LAUNCH_KERNEL(3424); break;
	case 3425: LAUNCH_KERNEL(3425); break;
	case 3426: LAUNCH_KERNEL(3426); break;
	case 3427: LAUNCH_KERNEL(3427); break;
	case 3428: LAUNCH_KERNEL(3428); break;
	case 3429: LAUNCH_KERNEL(3429); break;
	case 3430: LAUNCH_KERNEL(3430); break;
	case 3431: LAUNCH_KERNEL(3431); break;
	case 3432: LAUNCH_KERNEL(3432); break;
	case 3433: LAUNCH_KERNEL(3433); break;
	case 3434: LAUNCH_KERNEL(3434); break;
	case 3435: LAUNCH_KERNEL(3435); break;
	case 3436: LAUNCH_KERNEL(3436); break;
	case 3437: LAUNCH_KERNEL(3437); break;
	case 3438: LAUNCH_KERNEL(3438); break;
	case 3439: LAUNCH_KERNEL(3439); break;
	case 3440: LAUNCH_KERNEL(3440); break;
	case 3441: LAUNCH_KERNEL(3441); break;
	case 3442: LAUNCH_KERNEL(3442); break;
	case 3443: LAUNCH_KERNEL(3443); break;
	case 3444: LAUNCH_KERNEL(3444); break;
	case 3445: LAUNCH_KERNEL(3445); break;
	case 3446: LAUNCH_KERNEL(3446); break;
	case 3447: LAUNCH_KERNEL(3447); break;
	case 3448: LAUNCH_KERNEL(3448); break;
	case 3449: LAUNCH_KERNEL(3449); break;
	case 3450: LAUNCH_KERNEL(3450); break;
	case 3451: LAUNCH_KERNEL(3451); break;
	case 3452: LAUNCH_KERNEL(3452); break;
	case 3453: LAUNCH_KERNEL(3453); break;
	case 3454: LAUNCH_KERNEL(3454); break;
	case 3455: LAUNCH_KERNEL(3455); break;
	case 3456: LAUNCH_KERNEL(3456); break;
	case 3457: LAUNCH_KERNEL(3457); break;
	case 3458: LAUNCH_KERNEL(3458); break;
	case 3459: LAUNCH_KERNEL(3459); break;
	case 3460: LAUNCH_KERNEL(3460); break;
	case 3461: LAUNCH_KERNEL(3461); break;
	case 3462: LAUNCH_KERNEL(3462); break;
	case 3463: LAUNCH_KERNEL(3463); break;
	case 3464: LAUNCH_KERNEL(3464); break;
	case 3465: LAUNCH_KERNEL(3465); break;
	case 3466: LAUNCH_KERNEL(3466); break;
	case 3467: LAUNCH_KERNEL(3467); break;
	case 3468: LAUNCH_KERNEL(3468); break;
	case 3469: LAUNCH_KERNEL(3469); break;
	case 3470: LAUNCH_KERNEL(3470); break;
	case 3471: LAUNCH_KERNEL(3471); break;
	case 3472: LAUNCH_KERNEL(3472); break;
	case 3473: LAUNCH_KERNEL(3473); break;
	case 3474: LAUNCH_KERNEL(3474); break;
	case 3475: LAUNCH_KERNEL(3475); break;
	case 3476: LAUNCH_KERNEL(3476); break;
	case 3477: LAUNCH_KERNEL(3477); break;
	case 3478: LAUNCH_KERNEL(3478); break;
	case 3479: LAUNCH_KERNEL(3479); break;
	case 3480: LAUNCH_KERNEL(3480); break;
	case 3481: LAUNCH_KERNEL(3481); break;
	case 3482: LAUNCH_KERNEL(3482); break;
	case 3483: LAUNCH_KERNEL(3483); break;
	case 3484: LAUNCH_KERNEL(3484); break;
	case 3485: LAUNCH_KERNEL(3485); break;
	case 3486: LAUNCH_KERNEL(3486); break;
	case 3487: LAUNCH_KERNEL(3487); break;
	case 3488: LAUNCH_KERNEL(3488); break;
	case 3489: LAUNCH_KERNEL(3489); break;
	case 3490: LAUNCH_KERNEL(3490); break;
	case 3491: LAUNCH_KERNEL(3491); break;
	case 3492: LAUNCH_KERNEL(3492); break;
	case 3493: LAUNCH_KERNEL(3493); break;
	case 3494: LAUNCH_KERNEL(3494); break;
	case 3495: LAUNCH_KERNEL(3495); break;
	case 3496: LAUNCH_KERNEL(3496); break;
	case 3497: LAUNCH_KERNEL(3497); break;
	case 3498: LAUNCH_KERNEL(3498); break;
	case 3499: LAUNCH_KERNEL(3499); break;
	case 3500: LAUNCH_KERNEL(3500); break;
	case 3501: LAUNCH_KERNEL(3501); break;
	case 3502: LAUNCH_KERNEL(3502); break;
	case 3503: LAUNCH_KERNEL(3503); break;
	case 3504: LAUNCH_KERNEL(3504); break;
	case 3505: LAUNCH_KERNEL(3505); break;
	case 3506: LAUNCH_KERNEL(3506); break;
	case 3507: LAUNCH_KERNEL(3507); break;
	case 3508: LAUNCH_KERNEL(3508); break;
	case 3509: LAUNCH_KERNEL(3509); break;
	case 3510: LAUNCH_KERNEL(3510); break;
	case 3511: LAUNCH_KERNEL(3511); break;
	case 3512: LAUNCH_KERNEL(3512); break;
	case 3513: LAUNCH_KERNEL(3513); break;
	case 3514: LAUNCH_KERNEL(3514); break;
	case 3515: LAUNCH_KERNEL(3515); break;
	case 3516: LAUNCH_KERNEL(3516); break;
	case 3517: LAUNCH_KERNEL(3517); break;
	case 3518: LAUNCH_KERNEL(3518); break;
	case 3519: LAUNCH_KERNEL(3519); break;
	case 3520: LAUNCH_KERNEL(3520); break;
	case 3521: LAUNCH_KERNEL(3521); break;
	case 3522: LAUNCH_KERNEL(3522); break;
	case 3523: LAUNCH_KERNEL(3523); break;
	case 3524: LAUNCH_KERNEL(3524); break;
	case 3525: LAUNCH_KERNEL(3525); break;
	case 3526: LAUNCH_KERNEL(3526); break;
	case 3527: LAUNCH_KERNEL(3527); break;
	case 3528: LAUNCH_KERNEL(3528); break;
	case 3529: LAUNCH_KERNEL(3529); break;
	case 3530: LAUNCH_KERNEL(3530); break;
	case 3531: LAUNCH_KERNEL(3531); break;
	case 3532: LAUNCH_KERNEL(3532); break;
	case 3533: LAUNCH_KERNEL(3533); break;
	case 3534: LAUNCH_KERNEL(3534); break;
	case 3535: LAUNCH_KERNEL(3535); break;
	case 3536: LAUNCH_KERNEL(3536); break;
	case 3537: LAUNCH_KERNEL(3537); break;
	case 3538: LAUNCH_KERNEL(3538); break;
	case 3539: LAUNCH_KERNEL(3539); break;
	case 3540: LAUNCH_KERNEL(3540); break;
	case 3541: LAUNCH_KERNEL(3541); break;
	case 3542: LAUNCH_KERNEL(3542); break;
	case 3543: LAUNCH_KERNEL(3543); break;
	case 3544: LAUNCH_KERNEL(3544); break;
	case 3545: LAUNCH_KERNEL(3545); break;
	case 3546: LAUNCH_KERNEL(3546); break;
	case 3547: LAUNCH_KERNEL(3547); break;
	case 3548: LAUNCH_KERNEL(3548); break;
	case 3549: LAUNCH_KERNEL(3549); break;
	case 3550: LAUNCH_KERNEL(3550); break;
	case 3551: LAUNCH_KERNEL(3551); break;
	case 3552: LAUNCH_KERNEL(3552); break;
	case 3553: LAUNCH_KERNEL(3553); break;
	case 3554: LAUNCH_KERNEL(3554); break;
	case 3555: LAUNCH_KERNEL(3555); break;
	case 3556: LAUNCH_KERNEL(3556); break;
	case 3557: LAUNCH_KERNEL(3557); break;
	case 3558: LAUNCH_KERNEL(3558); break;
	case 3559: LAUNCH_KERNEL(3559); break;
	case 3560: LAUNCH_KERNEL(3560); break;
	case 3561: LAUNCH_KERNEL(3561); break;
	case 3562: LAUNCH_KERNEL(3562); break;
	case 3563: LAUNCH_KERNEL(3563); break;
	case 3564: LAUNCH_KERNEL(3564); break;
	case 3565: LAUNCH_KERNEL(3565); break;
	case 3566: LAUNCH_KERNEL(3566); break;
	case 3567: LAUNCH_KERNEL(3567); break;
	case 3568: LAUNCH_KERNEL(3568); break;
	case 3569: LAUNCH_KERNEL(3569); break;
	case 3570: LAUNCH_KERNEL(3570); break;
	case 3571: LAUNCH_KERNEL(3571); break;
	case 3572: LAUNCH_KERNEL(3572); break;
	case 3573: LAUNCH_KERNEL(3573); break;
	case 3574: LAUNCH_KERNEL(3574); break;
	case 3575: LAUNCH_KERNEL(3575); break;
	case 3576: LAUNCH_KERNEL(3576); break;
	case 3577: LAUNCH_KERNEL(3577); break;
	case 3578: LAUNCH_KERNEL(3578); break;
	case 3579: LAUNCH_KERNEL(3579); break;
	case 3580: LAUNCH_KERNEL(3580); break;
	case 3581: LAUNCH_KERNEL(3581); break;
	case 3582: LAUNCH_KERNEL(3582); break;
	case 3583: LAUNCH_KERNEL(3583); break;
	default: printf("intSalt: %d\n", intSalt); ASSERT(FALSE);
	}
}

#endif
